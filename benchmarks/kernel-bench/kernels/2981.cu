#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void tanh_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < size) {
        output[idx] = tanhf(input[idx]);
    }
}

torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    
    const int threads = 512;  // Changed from 256 to 512
    const int blocks = (input.numel() + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES(input.type(), "tanh_kernel", ([&] {
        tanh_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            input.numel()
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Tanh forward (CUDA)");
}