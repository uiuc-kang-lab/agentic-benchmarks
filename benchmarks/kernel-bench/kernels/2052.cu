#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32
#define VECTOR_SIZE 4
#define STRIDE_FACTOR 4  // Each thread processes multiple elements

__device__ __forceinline__ void load_tile_vectorized(const float* __restrict__ src,
                                                    float dst[TILE_SIZE][TILE_SIZE+1],
                                                    const int row, const int col,
                                                    const int N, const int stride) {
    float4 vec;
    int base_idx = row * stride + col;
    
    if (col + 4 <= N && row < N) {
        vec = *reinterpret_cast<const float4*>(&src[base_idx]);
        dst[threadIdx.y][threadIdx.x * 4] = vec.x;
        dst[threadIdx.y][threadIdx.x * 4 + 1] = vec.y;
        dst[threadIdx.y][threadIdx.x * 4 + 2] = vec.z;
        dst[threadIdx.y][threadIdx.x * 4 + 3] = vec.w;
    } else {
        #pragma unroll
        for (int i = 0; i < 4; i++) {
            dst[threadIdx.y][threadIdx.x * 4 + i] = (col + i < N && row < N) ? src[base_idx + i] : 0.0f;
        }
    }
}

__global__ void strided_vectorized_triangular_mm(const float* __restrict__ A,
                                                const float* __restrict__ B,
                                                float* __restrict__ C,
                                                const int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE+1];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE+1];

    // Base indices for this thread
    const int thread_row = blockIdx.y * TILE_SIZE + threadIdx.y;
    const int thread_col = blockIdx.x * TILE_SIZE + threadIdx.x * 4;

    // Process multiple elements per thread using strides
    #pragma unroll
    for (int s = 0; s < STRIDE_FACTOR; s++) {
        const int row = thread_row + s * (blockDim.y * gridDim.y);
        
        if (row >= N) continue;

        float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        
        // Handle upper triangular part
        if (row < thread_col && thread_col < N) {
            #pragma unroll
            for (int i = 0; i < 4; i++) {
                if (thread_col + i >= N) continue;
                C[row * N + thread_col + i] = 0.0f;
            }
            continue;
        }

        const int num_tiles = (N + TILE_SIZE - 1) / TILE_SIZE;
        
        for (int t = 0; t < num_tiles; t++) {
            const int tile_start = t * TILE_SIZE;
            if (tile_start > row) break;

            // Load tiles for current stride iteration
            load_tile_vectorized(A, As, row, tile_start + threadIdx.x * 4, N, N);
            load_tile_vectorized(B, Bs, tile_start + threadIdx.y, thread_col, N, N);
            
            __syncthreads();

            const int k_start = max(tile_start, thread_col);
            const int k_end = min(tile_start + TILE_SIZE, row + 1);

            #pragma unroll
            for (int k = k_start; k < k_end; k++) {
                const int bs_idx = k - tile_start;
                const float a_val = As[threadIdx.y][bs_idx];
                
                sum.x += a_val * Bs[bs_idx][threadIdx.x * 4];
                sum.y += a_val * Bs[bs_idx][threadIdx.x * 4 + 1];
                sum.z += a_val * Bs[bs_idx][threadIdx.x * 4 + 2];
                sum.w += a_val * Bs[bs_idx][threadIdx.x * 4 + 3];
            }
            
            __syncthreads();
        }

        // Store results for current stride
        if (row < N) {
            #pragma unroll
            for (int i = 0; i < 4; i++) {
                const int col = thread_col + i;
                if (col < N && row >= col) {
                    C[row * N + col] = ((float*)&sum)[i];
                }
            }
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Adjust grid size based on stride factor
    dim3 block(TILE_SIZE/4, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, 
              (N + TILE_SIZE * STRIDE_FACTOR - 1) / (TILE_SIZE * STRIDE_FACTOR));

    strided_vectorized_triangular_mm<<<grid, block>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Strided vectorized triangular matrix multiplication (CUDA)");
}