#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <cfloat>

// Optimized kernel using __ldg for read-only access and aligned memory ops
__global__ void ldg_aligned_argmax_kernel(
    const float* __restrict__ x,
    int64_t* __restrict__ indices,
    const int dimSize,
    const int innerSize) {

    // Each block handles one output element
    int global_idx = blockIdx.x;
    int outer_idx = global_idx / innerSize;
    int inner_idx = global_idx % innerSize;
    int base_offset = outer_idx * dimSize * innerSize + inner_idx;

    extern __shared__ float shared[];
    int* sidx = (int*)&shared[blockDim.x];

    float thread_max = -FLT_MAX;
    int thread_max_idx = 0;

    // Use __ldg for read-only cached access and aligned loads
    for (int i = threadIdx.x; i < dimSize; i += blockDim.x) {
        // 128-bit aligned access using __ldg intrinsic
        float val = __ldg(&x[base_offset + i * innerSize]);
        if (val > thread_max) {
            thread_max = val;
            thread_max_idx = i;
        }
    }

    shared[threadIdx.x] = thread_max;
    sidx[threadIdx.x] = thread_max_idx;
    __syncthreads();

    // Parallel reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            if (shared[threadIdx.x + s] > shared[threadIdx.x]) {
                shared[threadIdx.x] = shared[threadIdx.x + s];
                sidx[threadIdx.x] = sidx[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        indices[global_idx] = sidx[0];
    }
}

torch::Tensor ldg_aligned_argmax_forward_cuda(const torch::Tensor& x, const int64_t dim) {
    TORCH_CHECK(x.scalar_type() == at::kFloat, "Only float32 supported.");
    auto x_contig = x.contiguous();
    auto sizes = x_contig.sizes();
    int ndim = x_contig.dim();
    TORCH_CHECK(dim >= 0 && dim < ndim, "Invalid dim.");

    int outerSize = 1;
    for (int d = 0; d < dim; d++) outerSize *= sizes[d];
    int dimSize = sizes[dim];
    int innerSize = 1;
    for (int d = dim + 1; d < ndim; d++) innerSize *= sizes[d];

    std::vector<int64_t> out_sizes;
    for (int d = 0; d < ndim; d++) if (d != dim) out_sizes.push_back(sizes[d]);
    auto indices = torch::empty(out_sizes, torch::TensorOptions().device(x.device()).dtype(torch::kLong));

    // Use larger block size to maximize memory throughput
    int blockSize = 256;
    dim3 grid(outerSize * innerSize);
    size_t shared_mem = blockSize * (sizeof(float) + sizeof(int));

    ldg_aligned_argmax_kernel<<<grid, blockSize, shared_mem>>>(
        x_contig.data_ptr<float>(),
        indices.data_ptr<int64_t>(),
        dimSize,
        innerSize
    );

    return indices;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &ldg_aligned_argmax_forward_cuda, "LDG-aligned ArgMax CUDA forward");
}