#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Device helper: define an inline exponential function for float and double.
template <typename scalar_t>
__device__ inline scalar_t my_exp(scalar_t x);

template <>
__device__ inline float my_exp<float>(float x) {
    return expf(x);
}

template <>
__device__ inline double my_exp<double>(double x) {
    return exp(x);
}

// CUDA kernel implementing the SELU activation using a grid-stride loop to ensure
// memory coalescing. Each thread processes multiple elements with consecutive
// memory accesses, which improves the effective memory bandwidth on the GPU.

template <typename scalar_t>
__global__ void selu_kernel(const scalar_t* __restrict__ input,
                            scalar_t* __restrict__ output,
                            size_t numel) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
    // SELU parameters
    const scalar_t alpha = static_cast<scalar_t>(1.67326324235437728481);
    const scalar_t lambda = static_cast<scalar_t>(1.05070098735548049342);

    // Grid-stride loop to ensure threads in a warp access consecutive memory locations
    for (size_t i = idx; i < numel; i += stride) {
        scalar_t x = input[i];
        scalar_t value = (x > static_cast<scalar_t>(0)) 
                         ? x 
                         : alpha * (my_exp(x) - static_cast<scalar_t>(1));
        output[i] = lambda * value;
    }
}

// Host function that launches the CUDA SELU kernel.
// Exposed to Python as "forward" via pybind11.

torch::Tensor selu_forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    auto output = torch::empty_like(input);
    const size_t numel = input.numel();
    const int threads = 512;  // Increased thread count per block for better occupancy
    const int blocks = (numel + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "selu_forward_cuda", ([&] {
        const scalar_t *input_ptr = input.data_ptr<scalar_t>();
        scalar_t *output_ptr = output.data_ptr<scalar_t>();
        selu_kernel<scalar_t><<<blocks, threads>>>(input_ptr, output_ptr, numel);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &selu_forward, "SELU Activation Forward (CUDA)");
}
