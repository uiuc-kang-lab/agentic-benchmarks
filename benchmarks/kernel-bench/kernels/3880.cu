#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void softsign_kernel_optimized(const float* x, float* out, int num_elements) {
    extern __shared__ float shared_data[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int local_idx = threadIdx.x;

    // Load data into shared memory only if within bounds
    if (idx < num_elements) {
        shared_data[local_idx] = x[idx];
    }
    __syncthreads();

    // Perform the Softsign computation
    if (idx < num_elements) {
        float val = shared_data[local_idx];
        out[idx] = val / (1.0f + fabsf(val));
    }
}

torch::Tensor forward(torch::Tensor x) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int num_elements = x.numel();
    int threads = 1024;
    int blocks = (num_elements + threads - 1) / threads;

    size_t shared_memory_size = threads * sizeof(float);
    softsign_kernel_optimized<<<blocks, threads, shared_memory_size>>>(
        x.data_ptr<float>(), out.data_ptr<float>(), num_elements
    );

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Softsign activation (CUDA)");
}