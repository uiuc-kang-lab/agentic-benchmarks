#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Kernel optimized with stride loop and shared memory for better performance

template <typename scalar_t>
__global__ void cumprod_optimized_kernel(
    scalar_t* output,
    const scalar_t* input,
    const int64_t dim_size,
    const int64_t stride,
    const int64_t total_batches) {

    extern __shared__ scalar_t shared_data[];

    for (int idx = blockIdx.x * blockDim.x + threadIdx.x;
         idx < total_batches;
         idx += blockDim.x * gridDim.x) {
        
        int batch_idx = idx / stride;
        int in_idx = idx % stride;
        scalar_t product = 1;
        
        // Load input data to shared memory
        for (int i = threadIdx.x; i < dim_size; i += blockDim.x) {
            int64_t curr_idx = batch_idx * (dim_size * stride) + i * stride + in_idx;
            shared_data[i] = input[curr_idx];
        }
        __syncthreads();
        
        // Perform cumulative product using shared memory
        for (int i = 0; i < dim_size; i++) {
            product *= shared_data[i];
            int64_t curr_idx = batch_idx * (dim_size * stride) + i * stride + in_idx;
            output[curr_idx] = product;
        }
    }
}

torch::Tensor cumprod_cuda_optimized_forward(torch::Tensor input, int64_t dim) {
    // Create an empty output tensor with the same size as input
    auto output = torch::empty_like(input);
    
    // Retrieve tensor size and stride information
    auto sizes = input.sizes();
    auto strides = input.strides();
    
    // Dimension properties
    int64_t dim_size = sizes[dim];
    int64_t stride = strides[dim];
    
    // Total number of cumulative-product batches to process
    int64_t total_batches = input.numel() / dim_size;

    // Set CUDA kernel launch parameters
    const int threads = 256;
    const int blocks = (total_batches + threads - 1) / threads;
    const int shared_memory_size = dim_size * sizeof(scalar_t);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "cumprod_cuda_optimized", ([&] {
        cumprod_optimized_kernel<scalar_t><<<blocks, threads, shared_memory_size>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            dim_size,
            stride,
            total_batches
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &cumprod_cuda_optimized_forward, "Cumulative product forward (CUDA with optimized stride loop)");
}
