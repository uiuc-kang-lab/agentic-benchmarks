#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ATen/AccumulateType.h>

static const int NUM_STREAMS = 4;
static hipStream_t streams[NUM_STREAMS];
static bool streams_created = false;

void create_streams() {
    if (!streams_created) {
        for (int i = 0; i < NUM_STREAMS; i++) {
            hipStreamCreate(&streams[i]);
        }
        streams_created = true;
    }
}

void destroy_streams() {
    if (streams_created) {
        for (int i = 0; i < NUM_STREAMS; i++) {
            hipStreamDestroy(streams[i]);
        }
        streams_created = false;
    }
}

template <typename scalar_t>
__global__ void layernorm_streamed_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    const float eps,
    scalar_t* __restrict__ output,
    const int normalized_size,
    const int chunk_size,
    const int chunk_offset) {

    using accscalar_t = at::acc_type<scalar_t, true>;
    
    const int tidx = threadIdx.x;
    const int tidy = threadIdx.y;
    const int instance_idx = blockIdx.x + chunk_offset;
    
    extern __shared__ char smem[];
    accscalar_t* s_sum = reinterpret_cast<accscalar_t*>(smem);
    accscalar_t* s_sum_sq = s_sum + blockDim.x * blockDim.y;
    
    const scalar_t* in_ptr = input + instance_idx * normalized_size;
    scalar_t* out_ptr = output + instance_idx * normalized_size;
    
    const int thread_stride = blockDim.x * blockDim.y;
    const int thread_id = tidy * blockDim.x + tidx;
    
    accscalar_t local_sum = 0;
    accscalar_t local_sum_sq = 0;
    
    #pragma unroll 8
    for (int idx = thread_id; idx < normalized_size; idx += thread_stride) {
        accscalar_t val = static_cast<accscalar_t>(in_ptr[idx]);
        local_sum += val;
        local_sum_sq += val * val;
    }
    
    s_sum[thread_id] = local_sum;
    s_sum_sq[thread_id] = local_sum_sq;
    __syncthreads();
    
    if (thread_id < 32) {
        accscalar_t warp_sum = 0;
        accscalar_t warp_sum_sq = 0;
        
        #pragma unroll
        for (int i = thread_id; i < thread_stride; i += 32) {
            warp_sum += s_sum[i];
            warp_sum_sq += s_sum_sq[i];
        }
        
        #pragma unroll
        for (int offset = 16; offset > 0; offset /= 2) {
            warp_sum += __shfl_down_sync(0xffffffff, warp_sum, offset);
            warp_sum_sq += __shfl_down_sync(0xffffffff, warp_sum_sq, offset);
        }
        
        if (thread_id == 0) {
            s_sum[0] = warp_sum;
            s_sum_sq[0] = warp_sum_sq;
        }
    }
    __syncthreads();
    
    __shared__ accscalar_t mean, inv_std;
    if (thread_id == 0) {
        mean = s_sum[0] / normalized_size;
        accscalar_t variance = (s_sum_sq[0] / normalized_size) - (mean * mean);
        inv_std = rsqrt(variance + static_cast<accscalar_t>(eps));
    }
    __syncthreads();
    
    #pragma unroll 8
    for (int idx = thread_id; idx < normalized_size; idx += thread_stride) {
        accscalar_t val = static_cast<accscalar_t>(in_ptr[idx]);
        accscalar_t normalized = (val - mean) * inv_std;
        out_ptr[idx] = static_cast<scalar_t>(
            normalized * static_cast<accscalar_t>(weight[idx]) + 
            static_cast<accscalar_t>(bias[idx]));
    }
}

torch::Tensor layernorm_forward(torch::Tensor x, torch::Tensor weight, torch::Tensor bias, double eps = 1e-5) {
    create_streams();
    
    auto output = torch::empty_like(x);
    
    const int normalized_size = weight.numel();
    const int outer_size = x.numel() / normalized_size;
    const int chunk_size = (outer_size + NUM_STREAMS - 1) / NUM_STREAMS;
    
    const dim3 threads(32, 32);
    const int shared_mem_size = threads.x * threads.y * 2 * sizeof(float);
    
    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "layernorm_forward_cuda", ([&] {
        for (int i = 0; i < NUM_STREAMS; i++) {
            int stream_chunk_size = std::min(chunk_size, outer_size - i * chunk_size);
            if (stream_chunk_size <= 0) break;
            
            const dim3 blocks(stream_chunk_size);
            
            layernorm_streamed_kernel<scalar_t><<<blocks, threads, shared_mem_size, streams[i]>>>(
                x.data_ptr<scalar_t>(),
                weight.data_ptr<scalar_t>(),
                bias.data_ptr<scalar_t>(),
                static_cast<float>(eps),
                output.data_ptr<scalar_t>(),
                normalized_size,
                chunk_size,
                i * chunk_size);
        }
    }));
    
    // Synchronize all streams before returning
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &layernorm_forward, "LayerNorm forward (CUDA)",
          py::arg("x"), py::arg("weight"), py::arg("bias"), py::arg("eps") = 1e-5);
    // Add cleanup function for streams
    m.def("cleanup", &destroy_streams, "Cleanup CUDA streams");
}