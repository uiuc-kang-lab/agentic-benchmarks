#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Kernel that minimizes warp divergence by using warp-level ballot to decide fast (no-boundary-check) vs. safe paths.

__global__ void optimized_conv2d_branchless_warp_kernel(
    const float * __restrict__ input,
    const float * __restrict__ weight,
    const float * __restrict__ bias,
    float * __restrict__ output,
    int batch_size,
    int in_channels,
    int in_h,
    int in_w,
    int out_channels,
    int kernel_size,
    int out_h,
    int out_w,
    int stride,
    int padding) {

    // Determine output channel and batch index
    int oc = blockIdx.z % out_channels;
    int n  = blockIdx.z / out_channels;

    int out_row = blockIdx.y * blockDim.y + threadIdx.y;
    int out_col = blockIdx.x * blockDim.x + threadIdx.x;

    if (out_row >= out_h || out_col >= out_w) return;

    // Load filter weights for the current output channel into shared memory
    extern __shared__ float sh_weight[]; // size: in_channels * kernel_size * kernel_size
    int filter_elems = in_channels * kernel_size * kernel_size;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    for (int idx = tid; idx < filter_elems; idx += blockDim.x * blockDim.y) {
        int ic = idx / (kernel_size * kernel_size);
        int rem = idx % (kernel_size * kernel_size);
        int ki = rem / kernel_size;
        int kj = rem % kernel_size;
        // Weight layout assumed: [out_channels, in_channels, kernel_size, kernel_size]
        int weight_index = oc * filter_elems + ic * (kernel_size * kernel_size) + ki * kernel_size + kj;
        sh_weight[idx] = weight[weight_index];
    }
    __syncthreads();

    // Compute base input coordinate for this output pixel
    int base_in_row = out_row * stride - padding;
    int base_in_col = out_col * stride - padding;

    // Determine if the entire filter window lies in the input bounds
    bool interior = (base_in_row >= 0 && (base_in_row + kernel_size) <= in_h &&
                     base_in_col >= 0 && (base_in_col + kernel_size) <= in_w);

    // Use warp-level ballot to have a uniform decision within each warp
    unsigned int warp_mask = __ballot_sync(0xFFFFFFFF, interior);
    bool use_fast = (warp_mask == 0xFFFFFFFF);

    float sum = 0.0f;
    for (int ic = 0; ic < in_channels; ic++) {
        int input_channel_offset = n * (in_channels * in_h * in_w) + ic * (in_h * in_w);
        for (int ki = 0; ki < kernel_size; ki++) {
            for (int kj = 0; kj < kernel_size; kj++) {
                int filter_index = ic * (kernel_size * kernel_size) + ki * kernel_size + kj;
                if (use_fast) {
                    // Fast path: all threads are interior; no boundary check needed
                    int in_row = base_in_row + ki;
                    int in_col = base_in_col + kj;
                    int input_index = input_channel_offset + in_row * in_w + in_col;
                    sum += input[input_index] * sh_weight[filter_index];
                } else {
                    // Safe path: perform boundary check with a conditional operator
                    int in_row = base_in_row + ki;
                    int in_col = base_in_col + kj;
                    float val = ((in_row >= 0 && in_row < in_h && in_col >= 0 && in_col < in_w) ?
                                 input[input_channel_offset + in_row * in_w + in_col] : 0.0f);
                    sum += val * sh_weight[filter_index];
                }
            }
        }
    }

    if (bias != nullptr) {
        sum += bias[oc];
    }

    int out_index = n * (out_channels * out_h * out_w) + oc * (out_h * out_w) + out_row * out_w + out_col;
    output[out_index] = sum;
}


// Host function for forward pass
// Falls back to torch::conv2d for unsupported configurations

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    if (groups != 1 || dilation != 1) {
        if (bias.has_value()) {
            return torch::conv2d(x, weight, bias.value(), {stride, stride}, {padding, padding}, {dilation, dilation}, groups);
        } else {
            return torch::conv2d(x, weight, torch::Tensor(), {stride, stride}, {padding, padding}, {dilation, dilation}, groups);
        }
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int in_h = x.size(2);
    int in_w = x.size(3);
    int out_channels = weight.size(0);
    int kernel_size = weight.size(2); // square kernel assumed
    int out_h = (in_h + 2 * padding - kernel_size) / stride + 1;
    int out_w = (in_w + 2 * padding - kernel_size) / stride + 1;

    auto output = torch::empty({batch_size, out_channels, out_h, out_w}, x.options());

    dim3 block(32, 8);
    dim3 grid((out_w + block.x - 1) / block.x,
              (out_h + block.y - 1) / block.y,
              batch_size * out_channels);

    size_t shared_mem_size = in_channels * kernel_size * kernel_size * sizeof(float);

    optimized_conv2d_branchless_warp_kernel<<<grid, block, shared_mem_size>>>(
         x.data_ptr<float>(),
         weight.data_ptr<float>(),
         bias.has_value() ? bias.value().data_ptr<float>() : nullptr,
         output.data_ptr<float>(),
         batch_size,
         in_channels,
         in_h,
         in_w,
         out_channels,
         kernel_size,
         out_h,
         out_w,
         stride,
         padding);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed : ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized branchless CUDA forward function for 2D convolution with warp-level divergence minimization");
}
