#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// This kernel minimizes warp divergence by ensuring that all threads in a warp follow the same execution path.
// It uses a single loop to handle both the outer and inner dimensions, avoiding conditional logic within the loop.

template <typename scalar_t>
__global__ void sum_reduce_kernel(
    const scalar_t* input,
    scalar_t* output,
    int64_t reduce_size,
    int64_t outer_size,
    int64_t inner_size) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    // Each thread processes multiple elements in a grid-stride loop
    for (int idx = tid; idx < outer_size * inner_size; idx += total_threads) {
        int outer_idx = idx / inner_size;
        int inner_idx = idx % inner_size;

        scalar_t sum = 0;
        int64_t base_idx = outer_idx * reduce_size * inner_size + inner_idx;

        // Perform reduction along the specified dimension
        for (int i = 0; i < reduce_size; i++) {
            sum += input[base_idx + i * inner_size];
        }

        output[outer_idx * inner_size + inner_idx] = sum;
    }
}

// CUDA wrapper
torch::Tensor sum_reduce_cuda(torch::Tensor input, int64_t dim) {
    // Handle negative dimensions
    if (dim < 0) dim += input.dim();

    auto sizes = input.sizes().vec();
    int64_t reduce_size = sizes[dim];

    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }

    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }

    // Set output size: the reduction dimension becomes 1
    sizes[dim] = 1;
    auto output = torch::empty(sizes, input.options());

    // Launch configuration: one block per outer index;
    // threads per block chosen to cover the inner dimension with a maximum of 1024 threads
    int threads = 1024;
    int blocks = (outer_size * inner_size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sum_reduce_cuda", ([&] {
        sum_reduce_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            reduce_size,
            outer_size,
            inner_size
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &sum_reduce_cuda, "Sum reduction forward (CUDA)");
}
