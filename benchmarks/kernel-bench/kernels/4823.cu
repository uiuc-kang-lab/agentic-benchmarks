#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void l1_norm_forward_kernel(const float* __restrict__ x,
                                       float* __restrict__ out,
                                       int N,
                                       int D) {
  extern __shared__ float sdata[];
  int row = blockIdx.x;
  float sum = 0.0f;

  // Accumulate partial sums of absolute values for this row
  for (int col = threadIdx.x; col < D; col += blockDim.x) {
    float val = x[row * D + col];
    sum += fabsf(val);
  }

  // Store partial sums in shared memory
  sdata[threadIdx.x] = sum;
  __syncthreads();

  // Reduce within the block
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (threadIdx.x < stride) {
      sdata[threadIdx.x] += sdata[threadIdx.x + stride];
    }
    __syncthreads();
  }

  // Avoid division by zero
  float total_sum = sdata[0];
  if (threadIdx.x == 0 && total_sum == 0.0f) {
    total_sum = 1e-12f;
    sdata[0] = total_sum;
  }
  __syncthreads();
  total_sum = sdata[0];

  // Normalize the row
  for (int col = threadIdx.x; col < D; col += blockDim.x) {
    out[row * D + col] = x[row * D + col] / total_sum;
  }
}

torch::Tensor forward(torch::Tensor x) {
  TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA.");
  TORCH_CHECK(x.dim() == 2, "Expected 2D tensor for this example.");
  x = x.contiguous();

  auto out = torch::empty_like(x);
  int N = x.size(0);
  int D = x.size(1);

  int threads = std::min<int>(1024, D);
  int shared_mem_size = threads * sizeof(float);

  l1_norm_forward_kernel<<<N, threads, shared_mem_size>>>(
    x.data_ptr<float>(),
    out.data_ptr<float>(),
    N,
    D
  );

  return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "L1 Normalization forward pass (CUDA)");
}