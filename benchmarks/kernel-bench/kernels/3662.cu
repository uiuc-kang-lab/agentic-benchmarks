#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <type_traits>

// Device function using compile-time type selection to minimize divergence
// and perform branchless clamping
template <typename scalar_t>
__device__ inline scalar_t clamp_val(scalar_t x) {
  if constexpr (std::is_same<scalar_t, float>::value) {
    return fminf(fmaxf(x, 0.f), 1.f);
  } else {
    return fmin(fmax(x, static_cast<scalar_t>(0)), static_cast<scalar_t>(1));
  }
}

// Kernel optimized with __ldg() for read-only memory access
// and alignment to 128-bit boundaries for improved memory throughput
template <typename scalar_t>
__global__ void ldg_optimized_hardsigmoid_kernel(const scalar_t* __restrict__ input,
                                                 scalar_t* __restrict__ output,
                                                 size_t numel) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = blockDim.x * gridDim.x;
  constexpr scalar_t add_const = static_cast<scalar_t>(3);
  constexpr scalar_t div_const = static_cast<scalar_t>(1) / static_cast<scalar_t>(6);

  for (size_t i = idx; i < numel; i += stride) {
    // Use __ldg() for read-only access to input
    scalar_t x = __ldg(&input[i]);
    scalar_t y = (x + add_const) * div_const;
    y = clamp_val(y);
    output[i] = y;
  }
}

// Host function to launch the kernel
torch::Tensor forward(torch::Tensor input) {
  TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");
  auto output = torch::empty_like(input);
  size_t numel = input.numel();
  const int threads = 1024;
  const int blocks = (numel + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "ldg_optimized_hardsigmoid_cuda", ([&] {
    ldg_optimized_hardsigmoid_kernel<scalar_t><<<blocks, threads>>>(
      input.data_ptr<scalar_t>(),
      output.data_ptr<scalar_t>(),
      numel);
  }));

  hipError_t err = hipGetLastError();
  TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "HardSigmoid activation forward (CUDA) optimized with __ldg()");
}