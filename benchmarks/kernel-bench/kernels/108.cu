#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Kernel for matrix multiplication using strided loops
__global__ void matrixMultiplyStrided(float *A, float *B, float *C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float Cvalue = 0.0f;

    // Stride loop to handle workloads larger than the number of available threads
    for (int i = row; i < M; i += blockDim.y * gridDim.y) {
        for (int j = col; j < N; j += blockDim.x * gridDim.x) {
            float sum = 0.0f;
            for (int k = 0; k < K; ++k) {
                sum += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}

void matrix_multiply_cuda(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    float *d_A = A.data_ptr<float>();
    float *d_B = B.data_ptr<float>();
    float *d_C = C.data_ptr<float>();

    // Define block size
    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    matrixMultiplyStrided<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, K);
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int N = B.size(1);

    // Create output tensor with same options as input
    auto options = torch::TensorOptions()
        .dtype(A.dtype())
        .device(A.device())
        .requires_grad(false);
    
    torch::Tensor C = torch::empty({M, N}, options);

    matrix_multiply_cuda(A, B, C);
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication using strided loops (CUDA)");
}