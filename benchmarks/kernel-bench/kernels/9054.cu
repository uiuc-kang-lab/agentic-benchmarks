#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Revised kernel: each block handles one (batch, out_channel) pair.
__global__ void conv1d_shared_memory_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* output,
    int B,
    int in_channels,
    int in_size,
    int out_channels,
    int kernel_size,
    int out_size,
    int stride,
    int dilation
) {
    extern __shared__ float shared_weight[];
    
    // Map block index to a (batch, out_channel) pair.
    int bc = blockIdx.x;  // bc in [0, B * out_channels)
    int b = bc / out_channels;
    int oc = bc % out_channels;

    // Each block loads the weight vector for its output channel into shared memory.
    int tid = threadIdx.x;
    int total_weights = in_channels * kernel_size;
    int load_per_thread = (total_weights + blockDim.x - 1) / blockDim.x;
    for (int i = 0; i < load_per_thread; i++) {
        int index = tid * load_per_thread + i;
        if (index < total_weights) {
            shared_weight[index] = weight[oc * total_weights + index];
        }
    }
    __syncthreads();

    // Each thread computes multiple output positions in this (b, oc) block.
    for (int o = tid; o < out_size; o += blockDim.x) {
        float sum = 0.0f;
        for (int ic = 0; ic < in_channels; ic++) {
            for (int k = 0; k < kernel_size; k++) {
                int input_pos = o * stride + k * dilation;
                if (input_pos < in_size) {
                    int x_idx = b * (in_channels * in_size) + ic * in_size + input_pos;
                    int w_idx = ic * kernel_size + k;
                    sum += x[x_idx] * shared_weight[w_idx];
                }
            }
        }
        if (bias != nullptr) {
            sum += bias[oc];
        }
        int out_idx = b * (out_channels * out_size) + oc * out_size + o;
        output[out_idx] = sum;
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int dilation
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(weight.dim() == 3, "weight must be 3D");
    TORCH_CHECK(weight.size(1) == x.size(1), "Input channels mismatch");

    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        TORCH_CHECK(bias->dim() == 1, "bias must be 1D");
        TORCH_CHECK(bias->size(0) == weight.size(0), "Bias size mismatch");
    }

    int B = x.size(0);
    int in_channels = x.size(1);
    int in_size = x.size(2);
    int out_channels = weight.size(0);
    int kernel_size = weight.size(2);

    int out_size = (in_size - dilation * (kernel_size - 1) - 1) / stride + 1;
    TORCH_CHECK(out_size > 0, "Invalid output size");

    auto output = torch::empty({B, out_channels, out_size}, x.options());

    if (output.numel() == 0) return output;

    const float* x_data = x.data_ptr<float>();
    const float* weight_data = weight.data_ptr<float>();
    const float* bias_data = bias ? bias->data_ptr<float>() : nullptr;
    float* output_data = output.data_ptr<float>();

    int threads = 256;
    int blocks = (B * out_channels * out_size + threads - 1) / threads;
    
    // Shared memory size for weights
    int shared_memory_size = in_channels * kernel_size * sizeof(float);

    conv1d_shared_memory_kernel<<<blocks, threads, shared_memory_size>>>(
        x_data,
        weight_data,
        bias_data,
        output_data,
        B,
        in_channels,
        in_size,
        out_channels,
        kernel_size,
        out_size,
        stride,
        dilation
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel launch error: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "1D convolution forward (CUDA)");
}