#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

#define TILE_SIZE 16

// CUDA kernel for computing C = A.T * B using optimized shared memory synchronization.
__global__ void matMulOptimizedSyncKernel(const float* __restrict__ A,
                                           const float* __restrict__ B,
                                           float* __restrict__ C,
                                           int K, int M, int N) {
    int row = blockIdx.x * TILE_SIZE + threadIdx.y;
    int col = blockIdx.y * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    int numTiles = (K + TILE_SIZE - 1) / TILE_SIZE;
    for (int t = 0; t < numTiles; t++) {
        int aIndex = t * TILE_SIZE + threadIdx.x;
        if (row < M && aIndex < K)
            tileA[threadIdx.y][threadIdx.x] = A[aIndex * M + row];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        int bIndex = t * TILE_SIZE + threadIdx.y;
        if (bIndex < K && col < N)
            tileB[threadIdx.y][threadIdx.x] = B[bIndex * N + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        #pragma unroll
        for (int k_inner = 0; k_inner < TILE_SIZE; k_inner++) {
            sum += tileA[threadIdx.y][k_inner] * tileB[k_inner][threadIdx.x];
        }

        // Synchronizing here to ensure the results of all computations are written before loading the next tile.
        __syncthreads();
    }

    if (row < M && col < N)
        C[row * N + col] = sum;
}

// The forward function exposed via PyBind11.
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "Input A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input B must be a CUDA tensor");
    TORCH_CHECK(A.dtype() == torch::kFloat32, "Input A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "Input B must be float32");

    int K = A.size(0);
    int M = A.size(1);
    TORCH_CHECK(B.size(0) == K, "Dimension mismatch: A and B must have the same first dimension (K)");
    int N = B.size(1);

    auto C = torch::zeros({M, N}, torch::device(A.device()).dtype(A.dtype()));

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((M + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    const float* A_ptr = A.data_ptr<float>();
    const float* B_ptr = B.data_ptr<float>();
    float* C_ptr = C.data_ptr<float>();

    matMulOptimizedSyncKernel<<<gridDim, blockDim>>>(A_ptr, B_ptr, C_ptr, K, M, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Compute C = A.T * B (CUDA) using optimized shared memory tiling with minimal synchronization");
}
