#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <c10/util/Optional.h>
#include <stdio.h>

// Compute greatest common divisor
__device__ int gcd(int a, int b) {
  while (b != 0) {
    int t = b;
    b = a % b;
    a = t;
  }
  return a;
}

// Compute minimum of two integers
__device__ int my_min(int a, int b) {
  return a < b ? a : b;
}

// CUDA kernel for 2D transposed convolution with manual loop unrolling for the inner channel loop
__global__ void conv_transpose2d_kernel_manual_unroll(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int batch,
    const int in_channels,
    const int in_h,
    const int in_w,
    const int out_channels,
    const int out_h,
    const int out_w,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_h,
    const int pad_w,
    const int dilation_h,
    const int dilation_w,
    const int groups,
    const int in_channels_per_group,
    const int out_channels_per_group) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch * out_channels * out_h * out_w;
  if (index >= total) return;

  // Decode flat index into (n, oc, oh, ow)
  int ow = index % out_w;
  int tmp = index / out_w;
  int oh = tmp % out_h;
  tmp = tmp / out_h;
  int oc = tmp % out_channels;
  int n = tmp / out_channels;

  // Initialize output value with bias
  float out_val = bias[oc];

  // Determine group index
  int g = oc / out_channels_per_group;

  // Precompute candidate positions (output coordinate plus padding)
  int candidate_h = oh + pad_h;
  int candidate_w = ow + pad_w;

  // Compute valid kernel offsets and steps for height dimension
  int offset_kh = -1;
  int mod_h = candidate_h % stride_h;
  for (int k = 0; k < stride_h; k++) {
    if ((k * dilation_h) % stride_h == mod_h) {
      offset_kh = k;
      break;
    }
  }
  int step_kh = stride_h / gcd(stride_h, dilation_h);
  int kh_bound = candidate_h / dilation_h + 1;
  int kh_end = my_min(kernel_h, kh_bound);

  // Compute valid kernel offsets and steps for width dimension
  int offset_kw = -1;
  int mod_w = candidate_w % stride_w;
  for (int k = 0; k < stride_w; k++) {
    if ((k * dilation_w) % stride_w == mod_w) {
      offset_kw = k;
      break;
    }
  }
  int step_kw = stride_w / gcd(stride_w, dilation_w);
  int kw_bound = candidate_w / dilation_w + 1;
  int kw_end = my_min(kernel_w, kw_bound);

  // Pre-calculate constants for input indexing
  int stride_c = in_h * in_w; // offset between consecutive channels in input
  int base_n = n * in_channels * stride_c;  // base offset for the nth example

  // Loop over kernel height and width with unrolling
  #pragma unroll
  for (int kh = offset_kh; kh < kh_end; kh += step_kh) {
    int h_in_candidate = candidate_h - kh * dilation_h;
    int ih = h_in_candidate / stride_h;
    if (ih < 0 || ih >= in_h) continue;

    #pragma unroll
    for (int kw = offset_kw; kw < kw_end; kw += step_kw) {
      int w_in_candidate = candidate_w - kw * dilation_w;
      int iw = w_in_candidate / stride_w;
      if (iw < 0 || iw >= in_w) continue;

      int offset_pixel = ih * in_w + iw;
      // Compute weight offset related constants
      int weight_channel_offset = (oc - g * out_channels_per_group) * (kernel_h * kernel_w) + kh * kernel_w + kw;
      int weight_stride = out_channels_per_group * kernel_h * kernel_w;

      // Manual unroll over the input channel loop for this group
      int group_start = g * in_channels_per_group;
      int group_end = group_start + in_channels_per_group;
      int c = group_start;
      int group_count = in_channels_per_group;
      int remainder = group_count % 4;
      int limit = group_end - remainder;

      for (; c < limit; c += 4) {
        int idx0 = base_n + c * stride_c + offset_pixel;
        int idx1 = base_n + (c + 1) * stride_c + offset_pixel;
        int idx2 = base_n + (c + 2) * stride_c + offset_pixel;
        int idx3 = base_n + (c + 3) * stride_c + offset_pixel;

        int w_idx0 = c * weight_stride + weight_channel_offset;
        int w_idx1 = (c + 1) * weight_stride + weight_channel_offset;
        int w_idx2 = (c + 2) * weight_stride + weight_channel_offset;
        int w_idx3 = (c + 3) * weight_stride + weight_channel_offset;

        out_val += x[idx0] * weight[w_idx0]
                 + x[idx1] * weight[w_idx1]
                 + x[idx2] * weight[w_idx2]
                 + x[idx3] * weight[w_idx3];
      }
      for (; c < group_end; c++) {
        int idx = base_n + c * stride_c + offset_pixel;
        int w_idx = c * weight_stride + weight_channel_offset;
        out_val += x[idx] * weight[w_idx];
      }
    }
  }

  int out_index = n * (out_channels * out_h * out_w) +
                  oc * (out_h * out_w) +
                  oh * out_w + ow;
  output[out_index] = out_val;
}

// Host function wrapper for the CUDA kernel
at::Tensor forward(
    at::Tensor x,
    at::Tensor weight,
    c10::optional<at::Tensor> bias,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> dilation,
    int groups) {
  x = x.contiguous();
  weight = weight.contiguous();
  if (bias.has_value() && bias.value().defined())
    bias = bias.value().contiguous();

  const int batch = x.size(0);
  const int in_channels = x.size(1);
  const int in_h = x.size(2);
  const int in_w = x.size(3);

  const int kernel_h = weight.size(2);
  const int kernel_w = weight.size(3);
  const int out_channels_per_group = weight.size(1);
  const int out_channels = out_channels_per_group * groups;

  const int stride_h = stride[0];
  const int stride_w = stride[1];
  const int pad_h = padding[0];
  const int pad_w = padding[1];
  const int dilation_h = dilation[0];
  const int dilation_w = dilation[1];

  const int out_h = (in_h - 1) * stride_h - 2 * pad_h + dilation_h * (kernel_h - 1) + 1;
  const int out_w = (in_w - 1) * stride_w - 2 * pad_w + dilation_w * (kernel_w - 1) + 1;

  if (!bias.has_value() || !bias.value().defined()) {
    bias = at::zeros({out_channels}, weight.options());
  }

  auto output = at::zeros({batch, out_channels, out_h, out_w}, x.options());

  int in_channels_per_group = in_channels / groups;

  int total_threads = batch * out_channels * out_h * out_w;
  const int threads = 256;
  const int blocks = (total_threads + threads - 1) / threads;

  conv_transpose2d_kernel_manual_unroll<<<blocks, threads>>>(
      x.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.value().data_ptr<float>(),
      output.data_ptr<float>(),
      batch, in_channels, in_h, in_w,
      out_channels, out_h, out_w,
      kernel_h, kernel_w,
      stride_h, stride_w,
      pad_h, pad_w,
      dilation_h, dilation_w,
      groups, in_channels_per_group, out_channels_per_group);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA kernel failed : %s\n", hipGetErrorString(err));
  }

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "2D Transposed Convolution with Manual Loop Unrolling (CUDA)",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride"),
        py::arg("padding"),
        py::arg("dilation"),
        py::arg("groups"));
}
