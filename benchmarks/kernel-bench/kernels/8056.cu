#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Kernel using shared memory for reduction
__global__ void conv_transposed_1d_shared_kernel(
    const float* __restrict__ input,   // [N, in_channels, input_width]
    const float* __restrict__ weight,  // [in_channels, out_channels, kernel_size]
    const float* __restrict__ bias,    // [out_channels] or nullptr
    float* __restrict__ output,        // [N, out_channels, output_width]
    int N,
    int in_channels,
    int out_channels,
    int input_width,
    int output_width,
    int kernel_size,
    int stride,
    int padding,
    int output_padding,
    int groups) {

    extern __shared__ float shared_data[];
    int tid = threadIdx.x;
    int ox = blockIdx.x * blockDim.x + tid;
    if (ox >= output_width) return;
    int oc = blockIdx.y;
    int n = blockIdx.z;

    int out_channels_per_group = out_channels / groups;
    int in_channels_per_group = in_channels / groups;
    int group = oc / out_channels_per_group;
    int ic_start = group * in_channels_per_group;

    float sum = (bias != nullptr) ? bias[oc] : 0.0f;

    for (int ic = 0; ic < in_channels_per_group; ic++) {
        int global_ic = ic_start + ic;
        for (int k = 0; k < kernel_size; k++) {
            int temp = ox + padding - k;
            if (temp < 0) continue;
            if ((temp % stride) != 0) continue;
            int ix = temp / stride;
            if (ix < 0 || ix >= input_width) continue;

            int input_index = n * (in_channels * input_width) + global_ic * input_width + ix;
            int weight_index = global_ic * (out_channels * kernel_size) + oc * kernel_size + k;
            sum += input[input_index] * weight[weight_index];
        }
    }

    shared_data[tid] = sum;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[n * (out_channels * output_width) + oc * output_width + ox] = shared_data[0];
    }
}

// Host function interfaced via pybind11 that prepares inputs and launches the kernel
torch::Tensor forward(
    const torch::Tensor& input,
    const torch::Tensor& weight,
    const c10::optional<torch::Tensor>& bias,
    int64_t stride,
    int64_t padding,
    int64_t output_padding,
    int64_t groups) {
    CHECK_INPUT(input);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    auto input_sizes = input.sizes();
    int N = input_sizes[0];
    int in_channels = input_sizes[1];
    int input_width = input_sizes[2];

    auto weight_sizes = weight.sizes();
    int out_channels = weight_sizes[1];
    int kernel_size = weight_sizes[2];

    int output_width = (input_width - 1) * stride - 2 * padding + kernel_size + output_padding;

    auto output = torch::zeros({N, out_channels, output_width}, input.options());

    const int threads = 256;
    const int blocks_x = (output_width + threads - 1) / threads;
    dim3 blocks(blocks_x, out_channels, N);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    size_t shared_mem_size = threads * sizeof(float);
    conv_transposed_1d_shared_kernel<<<blocks, threads, shared_mem_size, stream>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.has_value() ? bias.value().data_ptr<float>() : nullptr,
        output.data_ptr<float>(),
        N,
        in_channels,
        out_channels,
        input_width,
        output_width,
        kernel_size,
        stride,
        padding,
        output_padding,
        groups
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Transposed 1D convolution forward with shared memory (CUDA)");
}