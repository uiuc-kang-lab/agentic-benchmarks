#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 128
#define NUM_STREAMS 4

__global__ void triangular_mm_kernel_shared(const float* __restrict__ A,
                                             const float* __restrict__ B,
                                             float* __restrict__ C,
                                             int N,
                                             int tile_row) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Adjust row to account for tile offset
    row += tile_row;

    // Shared memory for tiles
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    float sum = 0.f;

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < N && t * TILE_SIZE + threadIdx.x < N) {
            As[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.f;
        }

        if (col < N && t * TILE_SIZE + threadIdx.y < N) {
            Bs[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.f;
        }

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = (row >= col) ? sum : 0.f;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Create CUDA streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);

    // Process matrix in tiles
    for (int tile_row = 0; tile_row < N; tile_row += TILE_SIZE) {
        int current_tile_size = min(TILE_SIZE, N - tile_row);
        dim3 numBlocks((N + threads - 1) / threads,
                      (current_tile_size + threads - 1) / threads);

        // Use stream based on current tile
        int stream_idx = (tile_row / TILE_SIZE) % NUM_STREAMS;
        
        triangular_mm_kernel_shared<<<numBlocks, threadsPerBlock, 0, streams[stream_idx]>>>(
            A.data_ptr<float>(),
            B.data_ptr<float>(),
            C.data_ptr<float>(),
            N,
            tile_row
        );
    }

    // Synchronize all streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}
