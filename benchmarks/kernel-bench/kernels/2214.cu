#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

// CUDA kernel for computing C = A.T * B with optimized memory access.
// A: shape (K, M), B: shape (K, N), C: shape (M, N)
__global__ void matMulKernelOptimized(const float* __restrict__ A,
                                      const float* __restrict__ B,
                                      float* __restrict__ C,
                                      int K,
                                      int M,
                                      int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < M && j < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; ++k) {
            sum += __ldg(&A[k * M + i]) * __ldg(&B[k * N + j]);
        }
        C[i * N + j] = sum;
    }
}

// The forward function exposed via PyBind11.
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "Input A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input B must be a CUDA tensor");
    TORCH_CHECK(A.dtype() == torch::kFloat32, "Input A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "Input B must be float32");

    int K = A.size(0);
    int M = A.size(1);
    TORCH_CHECK(B.size(0) == K, "Dimension mismatch: A and B must have the same first dimension (K)");
    int N = B.size(1);

    auto C = torch::zeros({M, N}, torch::device(A.device()).dtype(A.dtype()));

    const int THREADS = 16;
    dim3 blockDim(THREADS, THREADS);
    dim3 gridDim((M + THREADS - 1) / THREADS, (N + THREADS - 1) / THREADS);

    const float* A_ptr = A.data_ptr<float>();
    const float* B_ptr = B.data_ptr<float>();
    float* C_ptr = C.data_ptr<float>();

    matMulKernelOptimized<<<gridDim, blockDim>>>(A_ptr, B_ptr, C_ptr, K, M, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Compute C = A.T * B with optimized memory access (CUDA)");
}
