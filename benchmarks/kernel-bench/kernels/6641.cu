#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void max_reduce_opt_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t outer_size,
    const int64_t dim_size,
    const int64_t inner_size
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x; if (idx >= outer_size * inner_size) return;
    const int total_elements = outer_size * inner_size;
    
    if (idx >= total_elements) return;
    
    const int outer_idx = idx / inner_size;
    const int inner_idx = idx % inner_size;
    
    // Calculate starting position (coalesced pattern)
    const int64_t start_idx = outer_idx * dim_size * inner_size + inner_idx;
    
    // Initialize with first element (using read-only cache)
    scalar_t max_val = __ldg(input + start_idx);
    
    // Reduced reads via __ldg and sequential access pattern
    for (int i = 1; i < dim_size; i++) {
        const scalar_t val = __ldg(input + start_idx + i * inner_size);
        max_val = max(max_val, val);
    }
    
    output[idx] = max_val;
}

torch::Tensor max_reduce_cuda_forward(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();
    
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= input.size(i);
    }
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < input.dim(); i++) {
        inner_size *= input.size(i);
    }
    
    const int64_t dim_size = input.size(dim);
    
    auto output_sizes = input.sizes().vec();
    output_sizes.erase(output_sizes.begin() + dim);
    auto output = torch::empty(output_sizes, input.options());
    
    const int threads = 256;
    const int blocks = (outer_size * inner_size + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "max_reduce_forward", ([&] {
        max_reduce_opt_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            dim_size,
            inner_size
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_reduce_cuda_forward, "Max reduce forward (CUDA)");
}