#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel for KL divergence calculation with warp-level primitives
__global__ void kl_div_kernel_warp_optimized(
    const float* log_predictions,
    const float* targets, 
    float* output,
    const int n) {
    
    // Get global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    float sum = 0.0f;
    
    // Calculate KL divergence for this thread's elements
    for (int i = idx; i < n; i += blockDim.x * gridDim.x) {
        // F.kl_div implementation:
        // output = exp(log_predictions) - targets * log_predictions
        float log_pred = log_predictions[i];
        float target = targets[i];
        sum += expf(log_pred) - target * log_pred;
    }
    
    // Warp-level reduction using shuffle operations
    for (int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }
    
    // Write result for this warp to global memory
    if (threadIdx.x % 32 == 0) {
        atomicAdd(output, sum);
    }
}

torch::Tensor kl_div_cuda_forward_warp_optimized(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    // Get tensor sizes
    const int n = log_predictions.numel();
    
    // Create output tensor
    auto output = torch::zeros({1}, log_predictions.options());
    
    // Launch parameters
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;
    
    // Launch kernel
    kl_div_kernel_warp_optimized<<<blocks, threads>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward_warp_optimized, "KL divergence forward with warp optimization (CUDA)");
}