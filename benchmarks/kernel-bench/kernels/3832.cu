#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void softplus_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int unroll_factor = 4;
    
    // Process 4 elements per thread
    for (int idx = tid; idx < size; idx += stride * unroll_factor) {
        #pragma unroll
        for (int i = 0; i < unroll_factor; i++) {
            if (idx + i * stride < size) {
                const scalar_t x = input[idx + i * stride];
                // Reduce warp divergence by using math operations instead of branches
                const scalar_t exp_x = exp(x);
                const scalar_t is_large = x > 20.0;
                const scalar_t is_small = x < -20.0;
                output[idx + i * stride] = 
                    is_large * x +                    // Case x > 20.0
                    is_small * exp_x +                // Case x < -20.0
                    (!is_large && !is_small) * log1p(exp_x);  // Default case
            }
        }
    }
}

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    const int threads = 256;
    const int blocks = (size + threads * 4 - 1) / (threads * 4);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "softplus_forward_cuda", ([&] {
        softplus_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}