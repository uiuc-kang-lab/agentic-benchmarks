#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel that performs 1D max pooling. Each thread computes one output element.
// The kernel uses #pragma unroll to help the compiler optimize the loop over the pooling window.
__global__ void max_pool1d_tunable_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int64_t* __restrict__ indices,
    const int batch_size,
    const int num_channels,
    const int input_length,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation,
    const int output_length,
    const bool return_indices) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch_size * num_channels * output_length;
    if (tid >= total) return;

    // Decode the flattened index into batch, channel, and output index
    int o = tid % output_length;
    int tmp = tid / output_length;
    int c = tmp % num_channels;
    int b = tmp / num_channels;

    int input_start = o * stride - padding;
    float max_val = -INFINITY;
    int max_idx = -1;

    int base_idx = b * num_channels * input_length + c * input_length;

    #pragma unroll
    for (int k = 0; k < kernel_size; ++k) {
        int pos = input_start + k * dilation;
        if (pos >= 0 && pos < input_length) {
            float val = input[base_idx + pos];
            if (val > max_val) {
                max_val = val;
                max_idx = pos;
            }
        }
    }

    int out_idx = b * num_channels * output_length + c * output_length + o;
    output[out_idx] = max_val;
    if (return_indices) {
        indices[out_idx] = max_idx;
    }
}

// Host function that selects an optimal block size using the CUDA occupancy API
// and launches the kernel with that configuration.
torch::Tensor forward(
    torch::Tensor x,
    int64_t kernel_size,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    bool return_indices) {

    TORCH_CHECK(x.dim() == 3, "Input must be 3D");
    TORCH_CHECK(x.is_cuda(), "Input must be on CUDA");
    TORCH_CHECK(x.is_contiguous(), "Input must be contiguous");

    const int batch_size = x.size(0);
    const int num_channels = x.size(1);
    const int input_length = x.size(2);

    const int output_length = ((input_length + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    TORCH_CHECK(output_length > 0, "Output length must be positive");

    auto options = torch::TensorOptions().dtype(x.dtype()).device(x.device());
    auto output = torch::empty({batch_size, num_channels, output_length}, options);
    torch::Tensor indices;
    if (return_indices) {
        indices = torch::empty({batch_size, num_channels, output_length}, options.dtype(torch::kInt64));
    }

    int total_elements = batch_size * num_channels * output_length;

    // Use CUDA Occupancy API to select the optimal block size among candidates (e.g.,32, 64, 128, 256, 512)
    int minGridSize = 0;
    int optimalBlockSize = 0;
    hipError_t occErr = hipOccupancyMaxPotentialBlockSize(
        &minGridSize, 
        &optimalBlockSize, 
        max_pool1d_tunable_kernel, 
        0, 
        total_elements);
    if (occErr != hipSuccess) {
        // Fallback in case of error
        optimalBlockSize = 256;
    }

    int numBlocks = (total_elements + optimalBlockSize - 1) / optimalBlockSize;

    max_pool1d_tunable_kernel<<<numBlocks, optimalBlockSize>>>(
        x.data_ptr<float>(),
        output.data_ptr<float>(),
        return_indices ? indices.data_ptr<int64_t>() : nullptr,
        batch_size,
        num_channels,
        input_length,
        kernel_size,
        stride,
        padding,
        dilation,
        output_length,
        return_indices);

    return return_indices ? torch::cat({output, indices}, -1) : output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "MaxPool1D forward with tunable block size (CUDA)");
}
