#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

__global__ void warp_shuffle_vectorized_kl_div(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets,
    float* __restrict__ output,
    const int n) {

    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x * 4;
    float4 sum = {0, 0, 0, 0};
    
    // Vectorized memory access with float4
    for (int i = tid * 4; i < n; i += stride) {
        if (i + 3 < n) { // Check bounds for last elements
            float4 log_pred = *reinterpret_cast<const float4*>(&log_predictions[i]);
            float4 target = *reinterpret_cast<const float4*>(&targets[i]);
            sum.x += expf(log_pred.x) - target.x * log_pred.x;
            sum.y += expf(log_pred.y) - target.y * log_pred.y;
            sum.z += expf(log_pred.z) - target.z * log_pred.z;
            sum.w += expf(log_pred.w) - target.w * log_pred.w;
        }
    }

    // Horizontal sum within thread
    float thread_sum = sum.x + sum.y + sum.z + sum.w;

    // Warp-level reduction using shuffle
    unsigned int mask = 0xffffffff;
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
        thread_sum += __shfl_down_sync(mask, thread_sum, offset);

    // Block-level reduction
    __shared__ float block_sum[32];
    if (threadIdx.x % 32 == 0)
        block_sum[threadIdx.x / 32] = thread_sum;
    __syncthreads();

    // Final reduction and atomic add
    if (threadIdx.x == 0) {
        float total = 0;
        int warps = blockDim.x / 32;  // warp size
        for (int i = 0; i < warps; i++)
            total += block_sum[i];
        atomicAdd(output, total);
    }
}

torch::Tensor kl_div_cuda_forward(
    const torch::Tensor& log_predictions,
    const torch::Tensor& targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());

    const int threads = 256;
    const int blocks = (n + 4 * threads - 1) / (4 * threads);

    // Launch kernel
    warp_shuffle_vectorized_kl_div<<<blocks, threads>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "Warp Shuffle Vectorized KL Divergence Forward");
}