#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <cstdint>

__constant__ float c_scalar;

__global__ void multiplySharedMemoryKernel(const float* __restrict__ A,
                                           float* __restrict__ C,
                                           int64_t size) {
    extern __shared__ float4 smem[];

    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int warp_id = threadIdx.x / 32;
    const int lane_id = threadIdx.x % 32;
    
    // Load 4 float4 elements per thread to shared memory
    #pragma unroll
    for(int i = 0; i < 4; ++i) {
        int load_idx = tid + i * blockDim.x * gridDim.x;
        if(load_idx * 4 < size) {
            smem[threadIdx.x + i * blockDim.x] = 
                reinterpret_cast<const float4*>(A)[load_idx];
        }
    }
    __syncthreads();

    // Process data from shared memory with warp-level optimization
    float val = ((float*)smem)[threadIdx.x * 4];
    #pragma unroll
    for(int i = 0; i < 4; ++i) {
        float element = __shfl_down_sync(0xffffffff, val, i);
        if(lane_id + i < 32) {
            ((float*)smem)[threadIdx.x * 4 + i] = element * c_scalar;
        }
    }
    __syncthreads();

    // Store results back to global memory
    #pragma unroll
    for(int i = 0; i < 4; ++i) {
        int store_idx = tid + i * blockDim.x * gridDim.x;
        if(store_idx * 4 < size) {
            reinterpret_cast<float4*>(C)[store_idx] = 
                smem[threadIdx.x + i * blockDim.x];
        }
    }
}

torch::Tensor forward(torch::Tensor A, float s) {
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor.");
    TORCH_CHECK(A.scalar_type() == torch::kFloat, "Input tensor A must be of type float.");

    auto C = torch::empty_like(A);
    int64_t size = A.numel();
    
    // Configure kernel
    const int threads = 256;
    const int blocks = (size + threads * 4 - 1) / (threads * 4);
    
    // Copy scalar to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(c_scalar), &s, sizeof(float));
    
    // Launch kernel with shared memory allocation
    multiplySharedMemoryKernel<<<blocks, threads, threads * sizeof(float4) * 4>>>(
        A.data_ptr<float>(),
        C.data_ptr<float>(),
        size
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix-scalar multiplication with shared memory and warp optimization");
}