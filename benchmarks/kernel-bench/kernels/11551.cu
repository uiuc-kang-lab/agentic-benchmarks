#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 256
#define MAX_BLOCKS 1024
#define FULL_MASK 0xffffffff

__device__ __forceinline__ float compute_kl_div(float log_pred, float target) {
    return __expf(log_pred) - target * log_pred;
}

__device__ __forceinline__ float warp_reduce_sum(float val) {
    #pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(FULL_MASK, val, offset);
    }
    return val;
}

__global__ void optimized_kl_div_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    extern __shared__ float shared_mem[];
    float* const shared_log_pred = shared_mem;
    float* const shared_targets = &shared_mem[TILE_SIZE];
    float* const partial_sums = &shared_mem[2 * TILE_SIZE];
    
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int warp_id = tid >> 5;
    const int lane_id = tid & 31;
    
    float thread_sum = 0.0f;
    
    for (int tile_start = bid * TILE_SIZE; tile_start < n; tile_start += gridDim.x * TILE_SIZE) {
        const int remaining = n - tile_start;
        const int tile_elements = min(TILE_SIZE, remaining);
        
        if (tid < tile_elements) {
            shared_log_pred[tid] = log_predictions[tile_start + tid];
            shared_targets[tid] = targets[tile_start + tid];
        }
        __syncthreads();
        
        #pragma unroll 4
        for (int i = tid; i < tile_elements; i += blockDim.x) {
            thread_sum += compute_kl_div(shared_log_pred[i], shared_targets[i]);
        }
        __syncthreads();
    }
    
    thread_sum = warp_reduce_sum(thread_sum);
    
    if (lane_id == 0) {
        partial_sums[warp_id] = thread_sum;
    }
    __syncthreads();
    
    if (warp_id == 0 && lane_id < (blockDim.x >> 5)) {
        float sum = partial_sums[lane_id];
        sum = warp_reduce_sum(sum);
        
        if (lane_id == 0) {
            atomicAdd(output, sum);
        }
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    const int threads = TILE_SIZE;
    const int blocks = min((n + TILE_SIZE - 1) / TILE_SIZE, MAX_BLOCKS);
    const int shared_mem = (2 * TILE_SIZE + threads/32) * sizeof(float);
    
    optimized_kl_div_kernel<<<blocks, threads, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward (CUDA)");
}