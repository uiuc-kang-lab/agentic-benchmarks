#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define UNROLL_FACTOR 8

__global__ void hinge_loss_kernel(const float* predictions, const float* targets, float* output, int n) {
    extern __shared__ float shared_mem[];
    float* shared_preds = shared_mem;
    float* shared_targets = &shared_mem[blockDim.x * UNROLL_FACTOR];
    
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int base_idx = (bid * blockDim.x + tid) * UNROLL_FACTOR;
    
    // Load data into shared memory
    #pragma unroll
    for (int i = 0; i < UNROLL_FACTOR; i++) {
        int idx = base_idx + i;
        if (idx < n) {
            shared_preds[tid * UNROLL_FACTOR + i] = predictions[idx];
            shared_targets[tid * UNROLL_FACTOR + i] = targets[idx];
        }
    }
    __syncthreads();
    
    // Process data from shared memory
    #pragma unroll
    for (int i = 0; i < UNROLL_FACTOR; i++) {
        int idx = base_idx + i;
        if (idx < n) {
            float pred = shared_preds[tid * UNROLL_FACTOR + i];
            float target = shared_targets[tid * UNROLL_FACTOR + i];
            float pred_target = pred * target;
            output[idx] = (pred_target < 1.0f) ? (1.0f - pred_target) : 0.0f;
        }
    }
}

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    CHECK_INPUT(predictions);
    CHECK_INPUT(targets);

    int n = predictions.numel();
    torch::Tensor output = torch::empty_like(predictions);

    int threads = 256;
    int elements_per_thread = UNROLL_FACTOR;
    int total_threads_needed = (n + elements_per_thread - 1) / elements_per_thread;
    int blocks = (total_threads_needed + threads - 1) / threads;
    blocks = min(blocks, 65535);

    hinge_loss_kernel<<<blocks, threads>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return torch::mean(output);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Hinge Loss Forward (Unrolled)");
}