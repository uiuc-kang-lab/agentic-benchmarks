#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Define constant memory for diagonal matrix A
__constant__ float const_A[1024];  // Assuming a maximum size of 1024 for demonstration

// CUDA kernel: each thread computes one element of the output C
// C[i, j] = A[i] * B[i, j] where A is diagonal
__global__ void diag_matmul_kernel(
    const float* __restrict__ B,
    float* __restrict__ C,
    const int64_t N,
    const int64_t M
) {
    // Calculate 2D thread index
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Each thread handles one row
    if (row < N) {
        // Get the diagonal element for this row
        const float diag_elem = const_A[row];
        
        // Multiply each element in the row by the diagonal element
        for (int col = 0; col < M; col++) {
            C[row * M + col] = diag_elem * B[row * M + col];
        }
    }
}

// Forward function that wraps our CUDA kernel
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.dim() == 1, "A must be a 1D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == B.size(0),
                "Dimension mismatch: A.size(0) must match B.size(0)");

    // Ensure inputs are on contiguous memory
    A = A.contiguous();
    B = B.contiguous();

    int64_t N = A.size(0);
    int64_t M = B.size(1);

    // Copy diagonal matrix A to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(const_A), A.data_ptr<float>(), N * sizeof(float));

    // Create an output tensor with the same device and type as B
    auto C = torch::empty({N, M}, B.options());

    // Configure and launch the kernel
    const int64_t threads = 256;
    const int64_t blocks = (N * M + threads - 1) / threads;
    diag_matmul_kernel<<<blocks, threads>>>(
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N,
        M
    );

    return C;
}

// Create the PyTorch extension module
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Diagonal matrix multiplication of A and B on the GPU");
}