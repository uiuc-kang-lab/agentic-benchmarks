#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_DIM 16

template <typename scalar_t>
__global__ void avg_pool2d_forward_shared_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int N,
    int C,
    int H,
    int W,
    int outH,
    int outW,
    int kernel_size,
    int stride,
    int padding
) {
    int w_out = blockIdx.x * blockDim.x + threadIdx.x;
    int h_out = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.z % C;
    int n = blockIdx.z / C;

    if (w_out < outW && h_out < outH && n < N) {
        int h_start = h_out * stride - padding;
        int w_start = w_out * stride - padding;
        
        scalar_t sum_val = 0;
        int count = 0;

        for (int kh = 0; kh < kernel_size; ++kh) {
            int h_in = h_start + kh;
            if (h_in >= 0 && h_in < H) {
                for (int kw = 0; kw < kernel_size; ++kw) {
                    int w_in = w_start + kw;
                    if (w_in >= 0 && w_in < W) {
                        sum_val += input[((n * C + c) * H + h_in) * W + w_in];
                        count++;
                    }
                }
            }
        }

        output[((n * C + c) * outH + h_out) * outW + w_out] = sum_val / static_cast<scalar_t>(count);
    }
}

torch::Tensor avg_pool2d_forward(
    torch::Tensor x,
    int kernel_size,
    int stride,
    int padding
) {
    TORCH_CHECK(x.dim() == 4, "Input must be a 4D tensor.");
    auto N = x.size(0);
    auto C = x.size(1);
    auto H = x.size(2);
    auto W = x.size(3);

    int outH = (H + 2 * padding - kernel_size) / stride + 1;
    int outW = (W + 2 * padding - kernel_size) / stride + 1;

    auto x_cont = x.contiguous();
    auto options = x.options();
    auto out = torch::empty({N, C, outH, outW}, options);

    dim3 threads(TILE_DIM, TILE_DIM);
    dim3 blocks((outW + TILE_DIM - 1) / TILE_DIM, (outH + TILE_DIM - 1) / TILE_DIM, C);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "avg_pool2d_forward_shared_kernel", ([&] {
        const scalar_t* input_data = x_cont.data_ptr<scalar_t>();
        scalar_t* output_data = out.data_ptr<scalar_t>();

        avg_pool2d_forward_shared_kernel<<<blocks, threads>>>(
            input_data,
            output_data,
            N, C, H, W,
            outH, outW,
            kernel_size, stride, padding
        );
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err));

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &avg_pool2d_forward, "2D Average Pooling forward using shared memory (CUDA)");
}