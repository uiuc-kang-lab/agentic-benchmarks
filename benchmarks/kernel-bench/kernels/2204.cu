#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

#define TILE_M 16
#define TILE_N 16
#define BLOCK_K 32

__global__ void hybridKernel(const float* __restrict__ A,
                            const float* __restrict__ B,
                            float* __restrict__ C,
                            int K, int M, int N) {
    int row = blockIdx.x * TILE_M + threadIdx.x;
    int col = blockIdx.y * TILE_N + threadIdx.y;

    float sum = 0.0f;

    __shared__ float As[BLOCK_K][TILE_M];
    __shared__ float Bs[BLOCK_K][TILE_N];

    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * blockDim.y;

    for (int k0 = 0; k0 < K; k0 += BLOCK_K) {
        for (int index = tid; index < BLOCK_K * TILE_M; index += totalThreads) {
            int t = index / TILE_M;
            int m = index % TILE_M;
            int global_k = k0 + t;
            int global_m = blockIdx.x * TILE_M + m;
            As[t][m] = (global_k < K && global_m < M) ? __ldg(&A[global_k * M + global_m]) : 0.0f;
        }

        for (int index = tid; index < BLOCK_K * TILE_N; index += totalThreads) {
            int t = index / TILE_N;
            int n = index % TILE_N;
            int global_k = k0 + t;
            int global_n = blockIdx.y * TILE_N + n;
            Bs[t][n] = (global_k < K && global_n < N) ? __ldg(&B[global_k * N + global_n]) : 0.0f;
        }

        __syncthreads();

        #pragma unroll
        for (int t = 0; t < BLOCK_K; t++) {
            sum += As[t][threadIdx.x] * Bs[t][threadIdx.y];
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "Input A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input B must be a CUDA tensor");
    TORCH_CHECK(A.dtype() == torch::kFloat32, "Input A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "Input B must be float32");

    int K = A.size(0);
    int M = A.size(1);
    TORCH_CHECK(B.size(0) == K, "Dimension mismatch: A and B must have the same first dimension (K)");
    int N = B.size(1);

    auto C = torch::zeros({M, N}, torch::device(A.device()).dtype(A.dtype()));

    dim3 block(TILE_M, TILE_N);
    dim3 grid((M + TILE_M - 1) / TILE_M, (N + TILE_N - 1) / TILE_N);

    const float* A_ptr = A.data_ptr<float>();
    const float* B_ptr = B.data_ptr<float>();
    float* C_ptr = C.data_ptr<float>();

    hybridKernel<<<grid, block>>>(A_ptr, B_ptr, C_ptr, K, M, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Compute C = A.T * B using a hybrid approach (CUDA)");
}
