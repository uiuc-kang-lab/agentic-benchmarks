#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

// CUDA kernel applying HardTanh activation
template <typename scalar_t>
__global__ void hardtanh_kernel(const scalar_t* __restrict__ x,
                                 scalar_t* __restrict__ out,
                                 int64_t numel,
                                 scalar_t min_val,
                                 scalar_t max_val) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numel) {
        scalar_t val = x[idx];
        // Branchless clamping operation
        val = fmaxf(val, min_val);
        val = fminf(val, max_val);
        out[idx] = val;
    }
}

at::Tensor forward_cuda(const at::Tensor& x, float min_val, float max_val) {
    auto out = at::empty_like(x);
    int64_t numel = x.numel();
    
    // Experimentally determined optimal block size for NVIDIA H100
    const int threads = 512;  
    const int blocks = (numel + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "hardtanh_cuda", ([&] {
        hardtanh_kernel<scalar_t><<<blocks, threads>>>(
            x.data_ptr<scalar_t>(),
            out.data_ptr<scalar_t>(),
            numel,
            static_cast<scalar_t>(min_val),
            static_cast<scalar_t>(max_val)
        );
    }));
    
    return out;
}

at::Tensor forward(const at::Tensor& x, float min_val, float max_val) {
    if (!x.is_cuda()) {
        throw std::invalid_argument("Input tensor must be a CUDA tensor");
    }
    return forward_cuda(x, min_val, max_val);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "HardTanh activation (CUDA) with block size 512");
}
