#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void hinge_loss_kernel(const float* predictions, const float* targets, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = fmaxf(0.0f, 1.0f - predictions[idx] * targets[idx]);
    }
}

__global__ void reduce_sum_kernel(const float* input, float* output, int n) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;
    if (idx < n) {
        sum = input[idx];
    }
    sdata[tid] = sum;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(output, sdata[0]);
    }
}

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    CHECK_INPUT(predictions);
    CHECK_INPUT(targets);

    int n = predictions.numel();
    torch::Tensor output = torch::empty_like(predictions);

    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    hinge_loss_kernel<<<blocks, threads>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    torch::Tensor sum_output = torch::zeros({1}, predictions.options());
    reduce_sum_kernel<<<blocks, threads, threads * sizeof(float)>>>(
        output.data_ptr<float>(),
        sum_output.data_ptr<float>(),
        n
    );

    return sum_output / n;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Hinge Loss Forward");
}