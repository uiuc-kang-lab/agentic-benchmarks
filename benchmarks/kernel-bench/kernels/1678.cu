#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   const int N) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row >= N || col >= N) return;
    
    if (row >= col) {
        float sum = 0.0f;
        
        // Process elements in tiles of 4
        const int TILE_SIZE = 4;
        int k = col;
        
        // Main loop with manual unrolling for tiles of 4
        #pragma unroll 1
        for (; k <= row - (TILE_SIZE - 1); k += TILE_SIZE) {
            // Manual unroll of 4 iterations
            sum += A[row * N + k] * B[k * N + col];
            sum += A[row * N + (k+1)] * B[(k+1) * N + col];
            sum += A[row * N + (k+2)] * B[(k+2) * N + col];
            sum += A[row * N + (k+3)] * B[(k+3) * N + col];
        }
        
        // Handle remaining elements
        #pragma unroll
        for (; k <= row; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        
        C[row * N + col] = sum;
    } else {
        C[row * N + col] = 0.0f;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Use 32x32 thread blocks to maximize occupancy
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}