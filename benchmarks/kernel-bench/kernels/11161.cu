#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// CUDA kernel for Smooth L1 Loss (Huber Loss) with stride loops
__global__ void smooth_l1_loss_kernel_stride(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    int n_elements
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    float thread_sum = 0.0f;

    // Stride loop to handle workloads larger than the number of threads
    for (int i = idx; i < n_elements; i += stride) {
        float diff = predictions[i] - targets[i];
        float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;
    }

    // Reduction in shared memory
    __shared__ float shared_sum[256];
    int tid = threadIdx.x;
    shared_sum[tid] = thread_sum;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_sum[tid] += shared_sum[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(output, shared_sum[0] / n_elements);
    }
}

// Host function wrapper
torch::Tensor smooth_l1_loss_cuda_stride(
    torch::Tensor predictions,
    torch::Tensor targets
) {
    TORCH_CHECK(
        predictions.sizes() == targets.sizes(),
        "Input tensors must have the same shape"
    );
    TORCH_CHECK(
        predictions.is_contiguous() && targets.is_contiguous(),
        "Input tensors must be contiguous"
    );
    TORCH_CHECK(
        predictions.device().is_cuda() && targets.device().is_cuda(),
        "Inputs must be CUDA tensors"
    );

    int n = predictions.numel();
    auto output = torch::zeros({1}, predictions.options());

    const int block_size = 256;
    const int grid_size = (n + block_size - 1) / block_size;

    smooth_l1_loss_kernel_stride<<<grid_size, block_size>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &smooth_l1_loss_cuda_stride, "Smooth L1 Loss (CUDA) with stride loops");
}
