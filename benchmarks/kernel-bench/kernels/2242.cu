#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

// Define tile dimension and split factor for the K dimension
#define TILE_DIM 16
#define SPLIT_K 2

// Optimized CUDA kernel for computing C = A.T * B.
// This combines shared memory tiling and split-K parallelism.
__global__ void optimizedMatmulKernel(const float* __restrict__ A,
                                       const float* __restrict__ B,
                                       float* __restrict__ C,
                                       int K, int M, int N) {
    int block_k_size = (K + SPLIT_K - 1) / SPLIT_K; // Calculate the size of each block for K dimension
    int k_start = blockIdx.z * block_k_size;
    int k_end = min(k_start + block_k_size, K);

    int row = blockIdx.x * TILE_DIM + threadIdx.y;
    int col = blockIdx.y * TILE_DIM + threadIdx.x;

    float cValue = 0.0f;

    __shared__ float As[TILE_DIM][TILE_DIM];
    __shared__ float Bs[TILE_DIM][TILE_DIM];

    int local_k = k_end - k_start;
    int numTiles = (local_k + TILE_DIM - 1) / TILE_DIM;

    for (int t = 0; t < numTiles; t++) {
        int k_idx = t * TILE_DIM + threadIdx.x;
        int global_k = k_start + k_idx;
        As[threadIdx.y][threadIdx.x] = (row < M && global_k < k_end) ? A[global_k * M + row] : 0.0f;

        int k_idx_b = t * TILE_DIM + threadIdx.y;
        global_k = k_start + k_idx_b;
        Bs[threadIdx.y][threadIdx.x] = (col < N && global_k < k_end) ? B[global_k * N + col] : 0.0f;

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE_DIM; k++) {
            cValue += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        atomicAdd(&C[row * N + col], cValue);
    }
}

// The forward function exposed via PyBind11
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "Input A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input B must be a CUDA tensor");
    TORCH_CHECK(A.dtype() == torch::kFloat32, "Input A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "Input B must be float32");

    int K = A.size(0);
    int M = A.size(1);
    TORCH_CHECK(B.size(0) == K, "Dimension mismatch: A and B must have the same first dimension (K)");
    int N = B.size(1);

    auto C = torch::zeros({M, N}, torch::device(A.device()).dtype(A.dtype()));

    dim3 blockDim(TILE_DIM, TILE_DIM);
    dim3 gridDim((M + TILE_DIM - 1) / TILE_DIM,
                 (N + TILE_DIM - 1) / TILE_DIM,
                 SPLIT_K);

    const float* A_ptr = A.data_ptr<float>();
    const float* B_ptr = B.data_ptr<float>();
    float* C_ptr = C.data_ptr<float>();

    optimizedMatmulKernel<<<gridDim, blockDim>>>(A_ptr, B_ptr, C_ptr, K, M, N);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Compute C = A.T * B optimized with shared memory and split-K parallelism (CUDA)");
}
