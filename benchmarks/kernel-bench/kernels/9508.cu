#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// Optimized CUDA kernel with 3D grid mapping for output dimensions
__global__ void conv_transpose2d_forward_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

  // Map threads in 3D: x for output width, y for output height, z for combined batch and channel index
  int w_out = blockIdx.x * blockDim.x + threadIdx.x; // output width index
  int h_out = blockIdx.y * blockDim.y + threadIdx.y; // output height index
  int batch_channel_idx = blockIdx.z;                // combined batch and out_channel index

  int b = batch_channel_idx / out_channels;          // batch index
  int o = batch_channel_idx % out_channels;          // output channel index

  // Boundary check
  if (b >= batch_size || o >= out_channels || h_out >= out_height || w_out >= out_width)
    return;

  float out_val = bias[o];
  
  // Iterate over input channels and kernel spatial dimensions
  for (int c = 0; c < in_channels; ++c) {
    for (int p = 0; p < kernel_size; ++p) {
      int h_unscaled = h_out + padding - p * dilation;
      if (h_unscaled % stride != 0)
        continue;
      int h_in = h_unscaled / stride;
      if (h_in < 0 || h_in >= in_height)
        continue;
      
      for (int q = 0; q < kernel_size; ++q) {
        int w_unscaled = w_out + padding - q * dilation;
        if (w_unscaled % stride != 0)
          continue;
        int w_in = w_unscaled / stride;
        if (w_in < 0 || w_in >= in_width)
          continue;
        
        int input_idx = ((b * in_channels + c) * in_height + h_in) * in_width + w_in;
        int weight_idx = ((c * out_channels + o) * kernel_size + p) * kernel_size + q;
        out_val += input[input_idx] * weight[weight_idx];
      }
    }
  }

  int output_idx = ((b * out_channels + o) * out_height + h_out) * out_width + w_out;
  output[output_idx] = out_val;
}

// CUDA launcher function
torch::Tensor conv_transpose2d_forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {
  
  // Input dimensions
  int batch_size = input.size(0);
  int in_channels = input.size(1);
  int in_height = input.size(2);
  int in_width  = input.size(3);
  
  // Weight dimensions: [in_channels, out_channels, kernel_size, kernel_size]
  int out_channels = weight.size(1);
  int kernel_size = weight.size(2); // assuming square kernel
  
  // Compute output dimensions
  int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  int out_width  = (in_width  - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  
  auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());

  // Use 3D grid: x for width, y for height, z for combined batch and channel
  dim3 block(16, 16);
  dim3 grid((out_width + block.x - 1) / block.x,
            (out_height + block.y - 1) / block.y,
            batch_size * out_channels);

  conv_transpose2d_forward_kernel<<<grid, block>>>(
      input.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      output.data_ptr<float>(),
      batch_size,
      in_channels,
      out_channels,
      in_height,
      in_width,
      kernel_size,
      out_height,
      out_width,
      stride,
      padding,
      dilation);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in conv_transpose2d_forward_kernel: %s\n", hipGetErrorString(err));
  }
  
  return output;
}

// Wrapper function to handle bias being None
torch::Tensor conv_transpose2d_forward_wrapper(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,  // accepts None
    int stride,
    int padding,
    int dilation) {
  
  int out_channels = weight.size(1);
  torch::Tensor bias;
  if (bias_obj.is(pybind11::none())) {
    bias = torch::zeros({out_channels}, weight.options());
  } else {
    bias = bias_obj.cast<torch::Tensor>();
  }
  
  return conv_transpose2d_forward_cuda(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &conv_transpose2d_forward_wrapper,
        "ConvTranspose2d forward (CUDA) with 3D grid indexing",
        pybind11::arg("input"),
        pybind11::arg("weight"),
        pybind11::arg("bias"),
        pybind11::arg("stride"),
        pybind11::arg("padding"),
        pybind11::arg("dilation"));
}
