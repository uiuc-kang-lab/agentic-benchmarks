#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__constant__ int d_dim_size;
__constant__ int d_stride;

__global__ void prod_reduce_kernel(const float* input, float* output, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        float product = 1.0f;
        for (int i = 0; i < d_dim_size; ++i) {
            product *= input[idx + i * d_stride];
        }
        output[idx] = product;
    }
}

torch::Tensor forward(torch::Tensor x, int dim) {
    CHECK_INPUT(x);

    auto sizes = x.sizes().vec();
    int dim_size = sizes[dim];
    sizes.erase(sizes.begin() + dim);
    torch::Tensor output = torch::empty(sizes, x.options());

    int num_elements = output.numel();
    int stride = x.stride(dim);

    // Copy constants to device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_dim_size), &dim_size, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_stride), &stride, sizeof(int));

    const float* input_ptr = x.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();

    int threads = 1024;
    int blocks = (num_elements + threads - 1) / threads;

    prod_reduce_kernel<<<blocks, threads>>>(input_ptr, output_ptr, num_elements);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Product reduction over a dimension (CUDA)");
}