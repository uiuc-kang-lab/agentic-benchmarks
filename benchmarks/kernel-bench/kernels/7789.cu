#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void conv2d_ldg_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,  // may be nullptr if not provided
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int in_height,
    int in_width,
    int out_channels,
    int kernel_height,
    int kernel_width,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation,
    int groups) {

    int w = blockIdx.x * blockDim.x + threadIdx.x;
    int h = blockIdx.y * blockDim.y + threadIdx.y;
    int oc = blockIdx.z;

    if (w < out_width && h < out_height && oc < out_channels) {
      for (int b = 0; b < batch_size; ++b) {
        float sum = 0.0f;
        int group_out_channels = out_channels / groups;
        int group = oc / group_out_channels;
        int in_channels_per_group = in_channels / groups;

        for (int c = 0; c < in_channels_per_group; ++c) {
          int input_channel = group * in_channels_per_group + c;
          for (int kh = 0; kh < kernel_height; ++kh) {
            for (int kw = 0; kw < kernel_width; ++kw) {
              int in_y = h * stride - padding + kh * dilation;
              int in_x = w * stride - padding + kw * dilation;
              if (in_y >= 0 && in_y < in_height && in_x >= 0 && in_x < in_width) {
                int input_idx = ((b * in_channels + input_channel) * in_height + in_y) * in_width + in_x;
                int weight_idx = (((oc * in_channels_per_group + c) * kernel_height) + kh) * kernel_width + kw;
                sum += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
              }
            }
          }
        }
        if (bias != nullptr) {
          sum += __ldg(&bias[oc]);
        }
        int output_idx = ((b * out_channels + oc) * out_height + h) * out_width + w;
        output[output_idx] = sum;
      }
    }
}

// The forward function sets up dimensions and launches the kernel with read-only cache optimizations
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
      CHECK_INPUT(bias.value());
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int in_height = x.size(2);
    int in_width = x.size(3);
    int out_channels = weight.size(0);
    int kernel_height = weight.size(2);
    int kernel_width = weight.size(3);

    int out_height = (in_height + 2 * padding - dilation * (kernel_height - 1) - 1) / stride + 1;
    int out_width = (in_width + 2 * padding - dilation * (kernel_width - 1) - 1) / stride + 1;

    auto options = x.options();
    auto output = torch::empty({batch_size, out_channels, out_height, out_width}, options);

    const float* input_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();
    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
      bias_ptr = bias.value().data_ptr<float>();
    }

    dim3 block_size(32, 16, 1);
    dim3 grid_size(
        (out_width + block_size.x - 1) / block_size.x,
        (out_height + block_size.y - 1) / block_size.y,
        out_channels  // one block per output channel
    );

    conv2d_ldg_kernel<<<grid_size, block_size>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch_size,
        in_channels,
        in_height,
        in_width,
        out_channels,
        kernel_height,
        kernel_width,
        out_height,
        out_width,
        stride,
        padding,
        dilation,
        groups
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA 2D Convolution with __ldg Optimizations");
}