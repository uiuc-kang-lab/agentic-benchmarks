#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>

// CUDA kernel with loop unrolling via #pragma unroll to reduce loop overhead

template <typename scalar_t>
__global__ void max_pool3d_forward_kernel_unrolled(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t* __restrict__ indices,
    const int batch_size,
    const int channels,
    const int input_d, const int input_h, const int input_w,
    const int output_d, const int output_h, const int output_w,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation) {

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total = batch_size * channels * output_d * output_h * output_w;
    if (idx >= total) return;

    const int w_out = idx % output_w;
    const int h_out = (idx / output_w) % output_h;
    const int d_out = (idx / (output_w * output_h)) % output_d;
    const int c = (idx / (output_w * output_h * output_d)) % channels;
    const int b = idx / (output_w * output_h * output_d * channels);

    const int d_start = d_out * stride - padding;
    const int h_start = h_out * stride - padding;
    const int w_start = w_out * stride - padding;
    
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    int max_index = -1;

    // Unroll the loops to reduce overhead
    #pragma unroll
    for (int k_d = 0; k_d < kernel_size; k_d++) {
        int d_in = d_start + k_d * dilation;
        if (d_in < 0 || d_in >= input_d) continue;

        #pragma unroll
        for (int k_h = 0; k_h < kernel_size; k_h++) {
            int h_in = h_start + k_h * dilation;
            if (h_in < 0 || h_in >= input_h) continue;

            #pragma unroll
            for (int k_w = 0; k_w < kernel_size; k_w++) {
                int w_in = w_start + k_w * dilation;
                if (w_in < 0 || w_in >= input_w) continue;

                int input_idx = ((b * channels + c) * input_d + d_in) * input_h * input_w +
                                h_in * input_w + w_in;
                scalar_t val = input[input_idx];

                if (val > max_val) {
                    max_val = val;
                    max_index = input_idx;
                }
            }
        }
    }

    output[idx] = max_val;
    if (indices != nullptr) {
        indices[idx] = max_index;
    }
}

torch::Tensor max_pool3d_cuda_forward_unrolled(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation,
    bool return_indices,
    bool ceil_mode) {

    auto input_sizes = input.sizes();
    const int batch_size = input_sizes[0];
    const int channels = input_sizes[1];
    const int input_d = input_sizes[2];
    const int input_h = input_sizes[3];
    const int input_w = input_sizes[4];

    const int output_d = ceil_mode ? 
        ceil((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    const int output_h = ceil_mode ?
        ceil((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    const int output_w = ceil_mode ?
        ceil((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);

    auto output = torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options());
    auto indices = return_indices ? 
        torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options().dtype(torch::kLong)) :
        torch::Tensor();

    const int threads = 256;
    const int blocks = (batch_size * channels * output_d * output_h * output_w + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool3d_forward_cuda_unrolled", ([&] {
        max_pool3d_forward_kernel_unrolled<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            return_indices ? indices.data_ptr<int64_t>() : nullptr,
            batch_size, channels,
            input_d, input_h, input_w,
            output_d, output_h, output_w,
            kernel_size, stride, padding, dilation);
    }));

    if (return_indices) {
        return torch::stack({output, indices}, 0);
    }
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool3d_cuda_forward_unrolled, "Max Pool 3D forward with loop unrolling (CUDA)");
}
