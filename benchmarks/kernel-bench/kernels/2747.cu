#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Experiment with block sizes: try 32, 64, 128, 256, 512 (here we use 256 as a starting point)
constexpr int BLOCK_SIZE = 256;

__global__ void leaky_relu_kernel(const float* __restrict__ x, float* __restrict__ out, float negative_slope, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (; idx < n; idx += stride) {
        float val = x[idx];
        out[idx] = (val > 0.0f) ? val : val * negative_slope;
    }
}

torch::Tensor leaky_relu_forward(torch::Tensor x, float negative_slope) {
    CHECK_INPUT(x);
    auto out = torch::empty_like(x);
    int n = x.numel();
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    leaky_relu_kernel<<<blocks, BLOCK_SIZE>>>(x.data_ptr<float>(), out.data_ptr<float>(), negative_slope, n);
    
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &leaky_relu_forward, "LeakyReLU forward (CUDA) with block size tuning");
}
