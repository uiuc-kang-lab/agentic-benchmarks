#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Warp-level reduction using shuffle intrinsics without extra synchronizations
__device__ __forceinline__ float warp_reduce(float val) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

// CUDA kernel for Smooth L1 Loss using vectorized loads and warp-level reduction
// This version avoids unnecessary __syncthreads() by letting each warp's leader perform an atomic add
// after reducing its lane values. This minimizes synchronization overhead.
__global__ void smooth_l1_loss_warp_atomic_kernel(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    const int n_elements
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    float thread_sum = 0.0f;

    // Process data in groups of 4 (vectorized access)
    int vec_count = n_elements / 4;
    const float4* pred4 = reinterpret_cast<const float4*>(predictions);
    const float4* targ4 = reinterpret_cast<const float4*>(targets);

    for (int i = idx; i < vec_count; i += stride) {
        float4 p = __ldg(pred4 + i);
        float4 t = __ldg(targ4 + i);
        
        float diff = p.x - t.x;
        float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;

        diff = p.y - t.y;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;
        
        diff = p.z - t.z;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;
        
        diff = p.w - t.w;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;
    }

    // Process any remaining elements with scalar loads
    int remainder_start = vec_count * 4;
    for (int i = remainder_start + idx; i < n_elements; i += stride) {
        float diff = __ldg(predictions + i) - __ldg(targets + i);
        float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;
    }

    // Perform warp-level reduction; no __syncthreads() needed within a warp
    int lane = threadIdx.x & (warpSize - 1);
    thread_sum = warp_reduce(thread_sum);

    // Each warp leader (lane 0) atomically adds its reduced sum to the global output
    if (lane == 0) {
        atomicAdd(output, thread_sum / n_elements);
    }
}

// Host function wrapping the CUDA kernel
torch::Tensor smooth_l1_loss_warp_atomic(
    torch::Tensor predictions,
    torch::Tensor targets
) {
    TORCH_CHECK(
        predictions.sizes() == targets.sizes(),
        "Input tensors must have the same shape"
    );
    TORCH_CHECK(
        predictions.is_contiguous() && targets.is_contiguous(),
        "Input tensors must be contiguous"
    );
    TORCH_CHECK(
        predictions.device().is_cuda() && targets.device().is_cuda(),
        "Inputs must be CUDA tensors"
    );

    int n = predictions.numel();
    auto output = torch::zeros({1}, predictions.options());

    const int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;

    smooth_l1_loss_warp_atomic_kernel<<<grid_size, block_size>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &smooth_l1_loss_warp_atomic, "Smooth L1 Loss (CUDA) with warp atomic reduction and minimal synchronization");
}
