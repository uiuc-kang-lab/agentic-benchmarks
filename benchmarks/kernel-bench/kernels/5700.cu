#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <limits>

// Kernel using grid-stride loops to handle workloads larger than available threads
// and ensure correct boundary handling

template <typename scalar_t, int KERNEL_SIZE>
__global__ void max_pool2d_stride_optim_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int stride,
    const int padding,
    const int dilation
) {
    const int total_elements = batch_size * channels * output_height * output_width;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int gridSize = blockDim.x * gridDim.x;

    while (index < total_elements) {
        int ow = index % output_width;
        int oh = (index / output_width) % output_height;
        int c = (index / (output_width * output_height)) % channels;
        int b = index / (output_width * output_height * channels);

        const int input_base = b * channels * input_height * input_width + 
                              c * input_height * input_width;

        scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

        #pragma unroll
        for (int kh = 0; kh < KERNEL_SIZE; kh++) {
            const int ih = oh * stride - padding + kh * dilation;
            if (ih >= 0 && ih < input_height) {
                #pragma unroll
                for (int kw = 0; kw < KERNEL_SIZE; kw++) {
                    const int iw = ow * stride - padding + kw * dilation;
                    if (iw >= 0 && iw < input_width) {
                        max_val = fmaxf(max_val, __ldg(&input[input_base + ih * input_width + iw]));
                    }
                }
            }
        }

        output[index] = max_val;
        index += gridSize;
    }
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const int output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const int output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int total_elements = output.numel();
    const int threads = 256;
    const int blocks = (total_elements + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool2d_cuda_forward", ([&] {
        switch(kernel_size) {
            case 2:
                max_pool2d_stride_optim_kernel<scalar_t, 2><<<blocks, threads>>>(
                    input.data_ptr<scalar_t>(),
                    output.data_ptr<scalar_t>(),
                    batch_size,
                    channels,
                    input_height,
                    input_width,
                    output_height,
                    output_width,
                    stride,
                    padding,
                    dilation
                );
                break;
            case 3:
                max_pool2d_stride_optim_kernel<scalar_t, 3><<<blocks, threads>>>(
                    input.data_ptr<scalar_t>(),
                    output.data_ptr<scalar_t>(),
                    batch_size,
                    channels,
                    input_height,
                    input_width,
                    output_height,
                    output_width,
                    stride,
                    padding,
                    dilation
                );
                break;
            default:
                max_pool2d_stride_optim_kernel<scalar_t, -1><<<blocks, threads>>>(
                    input.data_ptr<scalar_t>(),
                    output.data_ptr<scalar_t>(),
                    batch_size,
                    channels,
                    input_height,
                    input_width,
                    output_height,
                    output_width,
                    stride,
                    padding,
                    dilation
                );
        }
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Stride-optimized Max Pool 2D forward (CUDA)");
}
