#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void diag_matmul_kernel_2d(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    const int64_t N,
    const int64_t M
) {
    extern __shared__ float sA[];
    // 2D thread indexing
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < M) {
        const int idx = row * M + col;
        C[idx] = A[row] * B[idx];
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.dim() == 1, "A must be a 1D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == B.size(0), "Dimension mismatch: A.size(0) must match B.size(0)");

    A = A.contiguous();
    B = B.contiguous();

    int64_t N = A.size(0);
    int64_t M = B.size(1);

    auto C = torch::empty({N, M}, B.options());

    // 2D block configuration
    dim3 threads(16, 16);  // 256 threads per block in a 16x16 configuration
    dim3 blocks(
        (M + threads.x - 1) / threads.x,
        (N + threads.y - 1) / threads.y
    );

    diag_matmul_kernel_2d<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N,
        M
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Diagonal matrix multiplication using 2D grid");
}