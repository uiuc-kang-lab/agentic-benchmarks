#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK((x).is_cuda(), #x " must be a CUDA tensor");
#define CHECK_CONTIGUOUS(x) TORCH_CHECK((x).is_contiguous(), #x " must be contiguous");
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x);

template<int BLOCK_SIZE>
__global__ void transposed_conv3d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch,
    int in_channels,
    int in_d,
    int in_h,
    int in_w,
    int out_channels,
    int out_d,
    int out_h,
    int out_w,
    int k_d,
    int k_h,
    int k_w,
    int s_d,
    int s_h,
    int s_w,
    int p_d,
    int p_h,
    int p_w,
    int groups,
    int channels_per_group_in,
    int channels_per_group_out) {

    extern __shared__ float shared_mem[];

    const int total = batch * out_channels * out_d * out_h * out_w;
    
    for (int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x; idx < total; idx += BLOCK_SIZE * gridDim.x) {
        int tmp = idx;
        const int w_out = tmp % out_w; tmp /= out_w;
        const int h_out = tmp % out_h; tmp /= out_h;
        const int d_out = tmp % out_d; tmp /= out_d;
        const int oc = tmp % out_channels; tmp /= out_channels;
        const int n = tmp;

        float sum = (bias != nullptr) ? bias[oc] : 0.0f;

        const int group = oc / channels_per_group_out;
        const int oc_in_group = oc % channels_per_group_out;

        const int d_base = d_out + p_d;
        const int h_base = h_out + p_h;
        const int w_base = w_out + p_w;

        #pragma unroll 4
        for (int kd = 0; kd < k_d; kd++) {
            const int tmp_d = d_base - kd;
            if (tmp_d % s_d != 0) continue;
            const int in_d_idx = tmp_d / s_d;
            if (in_d_idx < 0 || in_d_idx >= in_d) continue;

            #pragma unroll 4
            for (int kh = 0; kh < k_h; kh++) {
                const int tmp_h = h_base - kh;
                if (tmp_h % s_h != 0) continue;
                const int in_h_idx = tmp_h / s_h;
                if (in_h_idx < 0 || in_h_idx >= in_h) continue;

                #pragma unroll 4
                for (int kw = 0; kw < k_w; kw++) {
                    const int tmp_w = w_base - kw;
                    if (tmp_w % s_w != 0) continue;
                    const int in_w_idx = tmp_w / s_w;
                    if (in_w_idx < 0 || in_w_idx >= in_w) continue;

                    #pragma unroll 4
                    for (int ic = 0; ic < channels_per_group_in; ic++) {
                        const int in_channel = group * channels_per_group_in + ic;
                        
                        const int input_idx = n * (in_channels * in_d * in_h * in_w) +
                                            in_channel * (in_d * in_h * in_w) +
                                            in_d_idx * (in_h * in_w) +
                                            in_h_idx * in_w + in_w_idx;
                        
                        const int weight_idx = in_channel * (channels_per_group_out * k_d * k_h * k_w) +
                                             oc_in_group * (k_d * k_h * k_w) +
                                             kd * (k_h * k_w) + kh * k_w + kw;
                        
                        sum += input[input_idx] * weight[weight_idx];
                    }
                }
            }
        }
        output[idx] = sum;
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias_opt,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> output_padding,
    int64_t groups) {

    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias_opt.has_value()) {
        CHECK_INPUT(*bias_opt);
    }

    const int batch = x.size(0);
    const int in_channels = x.size(1);
    const int in_d = x.size(2);
    const int in_h = x.size(3);
    const int in_w = x.size(4);

    const int k_d = weight.size(2);
    const int k_h = weight.size(3);
    const int k_w = weight.size(4);

    const int s_d = stride[0];
    const int s_h = stride[1];
    const int s_w = stride[2];
    const int p_d = padding[0];
    const int p_h = padding[1];
    const int p_w = padding[2];
    const int op_d = output_padding[0];
    const int op_h = output_padding[1];
    const int op_w = output_padding[2];

    const int out_d = (in_d - 1) * s_d - 2 * p_d + k_d + op_d;
    const int out_h = (in_h - 1) * s_h - 2 * p_h + k_h + op_h;
    const int out_w = (in_w - 1) * s_w - 2 * p_w + k_w + op_w;

    const int channels_per_group_out = weight.size(1);
    const int out_channels = channels_per_group_out * groups;
    const int channels_per_group_in = in_channels / groups;

    auto output = torch::zeros({batch, out_channels, out_d, out_h, out_w}, x.options());

    const float* x_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias_opt.has_value() ? (*bias_opt).data_ptr<float>() : nullptr;
    float* out_ptr = output.data_ptr<float>();

    const int total = batch * out_channels * out_d * out_h * out_w;
    
    int block_size;
    if (total <= 32768) block_size = 32;
    else if (total <= 65536) block_size = 64;
    else if (total <= 131072) block_size = 128;
    else if (total <= 262144) block_size = 256;
    else block_size = 512;

    const int num_blocks = (total + block_size - 1) / block_size;
    const int grid_size = min(num_blocks, 65535);

    switch(block_size) {
        case 32:
            transposed_conv3d_kernel<32><<<grid_size, 32>>>(x_ptr, weight_ptr, bias_ptr, out_ptr,
                batch, in_channels, in_d, in_h, in_w, out_channels, out_d, out_h, out_w,
                k_d, k_h, k_w, s_d, s_h, s_w, p_d, p_h, p_w, groups,
                channels_per_group_in, channels_per_group_out);
            break;
        case 64:
            transposed_conv3d_kernel<64><<<grid_size, 64>>>(x_ptr, weight_ptr, bias_ptr, out_ptr,
                batch, in_channels, in_d, in_h, in_w, out_channels, out_d, out_h, out_w,
                k_d, k_h, k_w, s_d, s_h, s_w, p_d, p_h, p_w, groups,
                channels_per_group_in, channels_per_group_out);
            break;
        case 128:
            transposed_conv3d_kernel<128><<<grid_size, 128>>>(x_ptr, weight_ptr, bias_ptr, out_ptr,
                batch, in_channels, in_d, in_h, in_w, out_channels, out_d, out_h, out_w,
                k_d, k_h, k_w, s_d, s_h, s_w, p_d, p_h, p_w, groups,
                channels_per_group_in, channels_per_group_out);
            break;
        case 256:
            transposed_conv3d_kernel<256><<<grid_size, 256>>>(x_ptr, weight_ptr, bias_ptr, out_ptr,
                batch, in_channels, in_d, in_h, in_w, out_channels, out_d, out_h, out_w,
                k_d, k_h, k_w, s_d, s_h, s_w, p_d, p_h, p_w, groups,
                channels_per_group_in, channels_per_group_out);
            break;
        default:
            transposed_conv3d_kernel<512><<<grid_size, 512>>>(x_ptr, weight_ptr, bias_ptr, out_ptr,
                batch, in_channels, in_d, in_h, in_w, out_channels, out_d, out_h, out_w,
                k_d, k_h, k_w, s_d, s_h, s_w, p_d, p_h, p_w, groups,
                channels_per_group_in, channels_per_group_out);
    }

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Dynamic Block Size Transposed Conv3D forward (CUDA)");
}