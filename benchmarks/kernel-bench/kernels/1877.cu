#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   const int N) {
    __shared__ float s_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float s_B[BLOCK_SIZE][BLOCK_SIZE];

    const int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    const int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    float sum = 0.0f;

    // Loop over blocks
    for (int bk = 0; bk <= (row / BLOCK_SIZE); bk++) {
        // Collaborative loading of A and B into shared memory
        const int block_start = bk * BLOCK_SIZE;
        
        if (row < N && (block_start + threadIdx.x) < N) {
            s_A[threadIdx.y][threadIdx.x] = A[row * N + block_start + threadIdx.x];
        } else {
            s_A[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        if ((block_start + threadIdx.y) < N && col < N) {
            s_B[threadIdx.y][threadIdx.x] = B[(block_start + threadIdx.y) * N + col];
        } else {
            s_B[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        __syncthreads();

        // Compute partial dot product using shared memory
        if (row < N && col < N && row >= col) {
            const int k_start = max(block_start, col);
            const int k_end = min(block_start + BLOCK_SIZE, row + 1);
            
            #pragma unroll 8
            for (int k = k_start; k < k_end; k++) {
                sum += s_A[threadIdx.y][k - block_start] * 
                       s_B[k - block_start][threadIdx.x];
            }
        }
        
        __syncthreads();
    }

    // Write result
    if (row < N && col < N) {
        if (row >= col) {
            C[row * N + col] = sum;
        } else {
            C[row * N + col] = 0.0f;
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    const int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
                   (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}