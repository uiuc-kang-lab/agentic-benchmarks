#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void conv1d_shared_memory_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* output,
    int B,
    int in_channels,
    int in_size,
    int out_channels,
    int kernel_size,
    int out_size,
    int stride,
    int dilation
) {
    extern __shared__ float shared_weight[];
    
    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + tid;
    const int total_elements = B * out_channels * out_size;
    
    if (idx < total_elements) {
        const int o = idx % out_size;
        const int tmp = idx / out_size;
        const int oc = tmp % out_channels;
        const int b = tmp / out_channels;

        // Load weights into shared memory
        const int weights_per_thread = (kernel_size * in_channels + blockDim.x - 1) / blockDim.x;
        for (int i = 0; i < weights_per_thread; i++) {
            const int weight_idx = tid * weights_per_thread + i;
            if (weight_idx < kernel_size * in_channels) {
                shared_weight[weight_idx] = weight[oc * (in_channels * kernel_size) + weight_idx];
            }
        }
        __syncthreads();

        float sum = 0.0f;
        for (int ic = 0; ic < in_channels; ++ic) {
            for (int k = 0; k < kernel_size; ++k) {
                const int input_pos = o * stride + k * dilation;
                if (input_pos < in_size) {
                    const int x_idx = b * (in_channels * in_size) + ic * in_size + input_pos;
                    const int w_idx = ic * kernel_size + k;
                    sum += x[x_idx] * shared_weight[w_idx];
                }
            }
        }

        if (bias != nullptr) {
            sum += bias[oc];
        }

        const int out_idx = b * (out_channels * out_size) + oc * out_size + o;
        output[out_idx] = sum;
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int dilation
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(weight.dim() == 3, "weight must be 3D");
    TORCH_CHECK(weight.size(1) == x.size(1), "Input channels mismatch");

    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        TORCH_CHECK(bias->dim() == 1, "bias must be 1D");
        TORCH_CHECK(bias->size(0) == weight.size(0), "Bias size mismatch");
    }

    int B = x.size(0);
    int in_channels = x.size(1);
    int in_size = x.size(2);
    int out_channels = weight.size(0);
    int kernel_size = weight.size(2);

    int out_size = (in_size - dilation * (kernel_size - 1) - 1) / stride + 1;
    TORCH_CHECK(out_size > 0, "Invalid output size");

    auto output = torch::empty({B, out_channels, out_size}, x.options());

    if (output.numel() == 0) return output;

    const float* x_data = x.data_ptr<float>();
    const float* weight_data = weight.data_ptr<float>();
    const float* bias_data = bias ? bias->data_ptr<float>() : nullptr;
    float* output_data = output.data_ptr<float>();

    int threads = 256;
    int blocks = (B * out_channels * out_size + threads - 1) / threads;
    
    // Shared memory size for weights
    int shared_memory_size = in_channels * kernel_size * sizeof(float);

    conv1d_shared_memory_kernel<<<blocks, threads, shared_memory_size>>>(
        x_data,
        weight_data,
        bias_data,
        output_data,
        B,
        in_channels,
        in_size,
        out_channels,
        kernel_size,
        out_size,
        stride,
        dilation
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel launch error: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "1D convolution forward (CUDA)");
}