#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void softplus_kernel_unrolled(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int num_threads = blockDim.x * gridDim.x;

    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        int idx = tid + i * num_threads;
        if (idx < size) {
            const scalar_t x = input[idx];
            if (x > 20.0) {
                output[idx] = x;
            } else if (x < -20.0) {
                output[idx] = exp(x);
            } else {
                output[idx] = log1p(exp(x));
            }
        }
    }
}

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    const int threads = 256;
    const int blocks = (size + 4 * threads - 1) / (4 * threads);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "softplus_forward_cuda", ([&] {
        softplus_kernel_unrolled<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}