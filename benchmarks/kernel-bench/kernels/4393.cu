#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Simple warp reduce sum for float
__inline__ __device__ float warpReduceSum(float val) {
#if defined(__CUDACC_VER_MAJOR__) && (__CUDACC_VER_MAJOR__ >= 9)
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
#else
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val += __shfl_down(val, offset);
    }
#endif
    return val;
}

// Block reduce sum using warp reduces
__inline__ __device__ float blockReduceSum(float val) {
    static __shared__ float shared[32]; // 1 warp per 32 threads
    int lane = threadIdx.x % warpSize;
    int wid  = threadIdx.x / warpSize;

    val = warpReduceSum(val);
    if (lane == 0) {
        shared[wid] = val;
    }
    __syncthreads();

    // Only the first warp is active here.
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0.0f;
    if (wid == 0) {
        val = warpReduceSum(val);
    }
    return val;
}

// CUDA kernel for instance normalization
__global__ void instance_norm_kernel(
    const float* __restrict__ x,
    float* __restrict__ y,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    int N,
    int C,
    int H,
    int W,
    float eps
) {
    int instance_id = blockIdx.x; // each block handles one (N, C) pair
    int n = instance_id / C;
    int c = instance_id % C;

    int HW = H * W;
    const float* x_ptr = x + (n * C + c) * HW;
    float* y_ptr       = y + (n * C + c) * HW;

    extern __shared__ float tile[];
    // Load input data into shared memory tile
    for (int i = threadIdx.x; i < HW; i += blockDim.x) {
        tile[i] = x_ptr[i];
    }
    __syncthreads();

    float sum_val = 0.0f, sum_sq_val = 0.0f;
    for (int i = threadIdx.x; i < HW; i += blockDim.x) {
        float v = tile[i];
        sum_val += v;
        sum_sq_val += v * v;
    }

    sum_val = blockReduceSum(sum_val);
    sum_sq_val = blockReduceSum(sum_sq_val);

    __shared__ float sharedMean;
    __shared__ float sharedVar;

    if (threadIdx.x == 0) {
        float mean = sum_val / (float)HW;
        float var = sum_sq_val / (float)HW - mean * mean;
        var = (var < 0.f) ? 0.f : var;
        sharedMean = mean;
        sharedVar = var;
    }
    __syncthreads();

    float mean = sharedMean;
    float var = sharedVar;
    float invert_std = 1.0f / sqrtf(var + eps);

    for (int i = threadIdx.x; i < HW; i += blockDim.x) {
        float v = x_ptr[i];
        float norm_val = (v - mean) * invert_std; // Pre-compute for memory efficiency
        norm_val = norm_val * (weight ? weight[c] : 1.0f) + (bias ? bias[c] : 0.0f);
        y_ptr[i] = norm_val;
    }
}

// Forward function called from Python
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    double eps
) {
    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    if (weight.defined()) {
        TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
    }
    if (bias.defined()) {
        TORCH_CHECK(bias.is_cuda(), "bias must be a CUDA tensor");
    }

    auto sizes = x.sizes();
    TORCH_CHECK(sizes.size() == 4, "Input tensor must be 4D: (N, C, H, W)");

    int N = sizes[0];
    int C = sizes[1];
    int H = sizes[2];
    int W = sizes[3];

    auto y = torch::empty_like(x);

    int threads = 256;
    int blocks = N * C;
    instance_norm_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        (weight.defined() && weight.numel() > 0) ? weight.data_ptr<float>() : nullptr,
        (bias.defined() && bias.numel() > 0) ? bias.data_ptr<float>() : nullptr,
        N,
        C,
        H,
        W,
        static_cast<float>(eps)
    );

    return y;
}

// Pybind11 module definition
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Instance Normalization forward (CUDA)");
}
