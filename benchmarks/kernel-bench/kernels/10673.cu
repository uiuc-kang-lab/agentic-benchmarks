#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Parallel warp-scan kernel for reverse cumulative sum along the last dimension.
// This kernel works for contiguous tensors along dimension (dim == ndim - 1) and when n <= 1024.
// Each block processes one row. The algorithm loads the row in reverse order, performs an inclusive scan using warp shuffle intrinsics,
// and then writes the result back in the correct order. Only two __syncthreads() calls are used (for cross-warp accumulation).

template <typename scalar_t>
__global__ void reverse_cumsum_parallel_kernel(const scalar_t* __restrict__ input,
                                                 scalar_t* __restrict__ output,
                                                 const int64_t n) {
    // Each block processes one row
    int row = blockIdx.x;
    const int64_t row_offset = row * n;

    const int tid = threadIdx.x;
    const int lane = tid & 31;       // lane index within a warp
    const int warp_id = tid >> 5;    // warp index within the block

    // Load element in reverse order if within bounds
    scalar_t val = 0;
    if (tid < n) {
        // Access reversed: thread tid loads element at index (n - 1 - tid)
        val = input[row_offset + (n - 1 - tid)];
    }

    // Intra-warp inclusive scan using warp shuffle
    // No __syncthreads() needed inside a warp
    for (int offset = 1; offset < 32; offset *= 2) {
        scalar_t n_val = __shfl_up_sync(0xffffffff, val, offset);
        if (lane >= offset) {
            val += n_val;
        }
    }

    // Write each warp's total (last active thread in the warp) into shared memory
    __shared__ scalar_t warp_sums[32];  // Maximum of 32 warps per block (1024 threads)
    __shared__ scalar_t warp_offsets[32];
    int block_warps = (n + 31) / 32;
    if ((lane == 31) || (tid == n - 1)) {
        warp_sums[warp_id] = val;
    }

    __syncthreads();

    // Compute the prefix sum of warp sums to get each warp's offset
    if (tid == 0) {
        warp_offsets[0] = 0;
        for (int i = 1; i < block_warps; i++) {
            warp_offsets[i] = warp_offsets[i - 1] + warp_sums[i - 1];
        }
    }

    __syncthreads();

    // Add the warp's offset to each thread's scan value
    if (tid < n && warp_id > 0) {
        val += warp_offsets[warp_id];
    }

    // Write the result back in reversed order to produce the reverse cumulative sum
    if (tid < n) {
        output[row_offset + (n - 1 - tid)] = val;
    }
}


// Sequential kernel fallback for non-contiguous cases or when n > 1024.
// This kernel uses a grid-stride loop to process each outer slice and does a sequential reverse cumulative sum.

template <typename scalar_t>
__global__ void reverse_cumsum_sequential_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t outer,
    const int64_t n,
    const int ndim,
    const int64_t* __restrict__ sizes,
    const int64_t* __restrict__ strides,
    const int dim) {

    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t gridStride = blockDim.x * gridDim.x;
    
    for (int64_t r = idx; r < outer; r += gridStride) {
        int64_t offset = 0;
        if (dim == ndim - 1) {
            offset = r * n;
        } else {
            int64_t tmp = r;
            for (int d = ndim - 1; d >= 0; d--) {
                if (d == dim) continue;
                int64_t cur_size = sizes[d];
                int64_t idx_d = tmp % cur_size;
                tmp /= cur_size;
                offset += idx_d * strides[d];
            }
        }
        int64_t stride_dim = (dim == ndim - 1) ? 1 : strides[dim];
        scalar_t cum = scalar_t(0);
        for (int64_t j = n - 1; j >= 0; j--) {
            int64_t cur_index = offset + j * stride_dim;
            cum += input[cur_index];
            output[cur_index] = cum;
        }
    }
}


at::Tensor reverse_cumsum(at::Tensor x, int64_t dim) {
    // Ensure the tensor is contiguous and on CUDA
    x = x.contiguous();
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");

    const int ndim = x.dim();
    TORCH_CHECK(dim >= 0 && dim < ndim, "dim out of range");

    auto output = at::empty_like(x);

    int64_t n = x.size(dim);
    int64_t outer = x.numel() / n;

    // Fast path: if the cumulative sum is along the last dimension and n <= 1024, use the parallel warp-scan kernel
    if (dim == ndim - 1 && n <= 1024) {
        // Determine block size as next power-of-2 >= n, up to 1024
        int threads = 1;
        while (threads < n) {
            threads *= 2;
        }
        if (threads > 1024) threads = 1024;

        // Each block processes one row
        dim3 blocks(outer);
        dim3 threadBlock(threads);

        AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "reverse_cumsum_cuda_parallel", ([&] {
            reverse_cumsum_parallel_kernel<scalar_t><<<blocks, threadBlock>>>(
                x.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                n);
        }));
    } else {
        // Fallback to sequential kernel
        // Prepare device arrays for sizes and strides
        const int max_dims = 10;
        int64_t h_sizes[max_dims], h_strides[max_dims];
        for (int i = 0; i < ndim; i++) {
            h_sizes[i] = x.size(i);
            h_strides[i] = x.stride(i);
        }

        int64_t *d_sizes = nullptr, *d_strides = nullptr;
        hipError_t err = hipMalloc(&d_sizes, ndim * sizeof(int64_t));
        TORCH_CHECK(err == hipSuccess, "hipMalloc for d_sizes failed");
        err = hipMalloc(&d_strides, ndim * sizeof(int64_t));
        TORCH_CHECK(err == hipSuccess, "hipMalloc for d_strides failed");

        err = hipMemcpy(d_sizes, h_sizes, ndim * sizeof(int64_t), hipMemcpyHostToDevice);
        TORCH_CHECK(err == hipSuccess, "hipMemcpy for d_sizes failed");
        err = hipMemcpy(d_strides, h_strides, ndim * sizeof(int64_t), hipMemcpyHostToDevice);
        TORCH_CHECK(err == hipSuccess, "hipMemcpy for d_strides failed");

        const int threads = 256;
        const int blocks = (outer + threads - 1) / threads;

        AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "reverse_cumsum_cuda_sequential", ([&] {
            reverse_cumsum_sequential_kernel<scalar_t><<<blocks, threads>>>(
                x.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                outer,
                n,
                ndim,
                d_sizes,
                d_strides,
                dim);
        }));

        hipFree(d_sizes);
        hipFree(d_strides);
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &reverse_cumsum, "Reverse cumulative sum with parallel warp-scan for contiguous tensors (CUDA)");
}
