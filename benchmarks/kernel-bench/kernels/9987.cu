#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void depthwise_conv2d_kernel(
    const float* __restrict__ input,
    const float* weight,
    const float* bias,
    float* output,
    int batch_size,
    int in_channels,
    int in_h,
    int in_w,
    int out_channels,
    int out_h,
    int out_w,
    int kernel_h,
    int kernel_w,
    int stride_h,
    int stride_w,
    int padding_h,
    int padding_w,
    int dilation_h,
    int dilation_w,
    int groups,
    int channels_per_group
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size * out_channels * out_h * out_w) return;

    int w_out = idx % out_w;
    idx /= out_w;
    int h_out = idx % out_h;
    idx /= out_h;
    int c_out = idx % out_channels;
    int b = idx / out_channels;

    if (b >= batch_size) return;

    int g = c_out / channels_per_group;
    int m = c_out % channels_per_group;

    float sum = 0.0f;
    
    #pragma unroll
    for (int kh = 0; kh < kernel_h; ++kh) {
        #pragma unroll
        for (int kw = 0; kw < kernel_w; ++kw) {
            int h_in = h_out * stride_h - padding_h + kh * dilation_h;
            int w_in = w_out * stride_w - padding_w + kw * dilation_w;
            
            if (h_in >= 0 && h_in < in_h && w_in >= 0 && w_in < in_w) {
                int input_idx = ((b * in_channels + g) * in_h + h_in) * in_w + w_in;
                int weight_idx = ((g * channels_per_group + m) * kernel_h + kh) * kernel_w + kw;
                sum += input[input_idx] * weight[weight_idx];
            }
        }
    }

    if (bias != nullptr) {
        sum += bias[c_out];
    }
    
    output[((b * out_channels + c_out) * out_h + h_out) * out_w + w_out] = sum;
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    int stride_h,
    int stride_w,
    int padding_h,
    int padding_w,
    int dilation_h,
    int dilation_w,
    int groups
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int in_h = x.size(2);
    int in_w = x.size(3);

    int kernel_h = weight.size(2);
    int kernel_w = weight.size(3);
    int out_channels = groups * weight.size(1);
    int channels_per_group = out_channels / groups;

    int out_h = (in_h + 2 * padding_h - dilation_h * (kernel_h - 1) - 1) / stride_h + 1;
    int out_w = (in_w + 2 * padding_w - dilation_w * (kernel_w - 1) - 1) / stride_w + 1;

    auto output = torch::empty({batch_size, out_channels, out_h, out_w}, x.options());

    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        bias_ptr = bias->data_ptr<float>();
    }

    int total_elements = batch_size * out_channels * out_h * out_w;
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;

    depthwise_conv2d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        in_h,
        in_w,
        out_channels,
        out_h,
        out_w,
        kernel_h,
        kernel_w,
        stride_h,
        stride_w,
        padding_h,
        padding_w,
        dilation_h,
        dilation_w,
        groups,
        channels_per_group
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Depthwise Conv2D forward (CUDA)");
}