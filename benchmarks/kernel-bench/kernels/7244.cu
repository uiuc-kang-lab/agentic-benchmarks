#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void conv2d_kernel(float* x, float* weight, float* bias, float* output, 
                              int stride, int padding, int dilation, int groups,
                              int x_height, int x_width, int weight_height, int weight_width) {
    const int batch_size = blockIdx.z;
    const int output_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int output_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int output_height = (x_height - weight_height + 2 * padding) / stride + 1;
    const int output_width = (x_width - weight_width + 2 * padding) / stride + 1;
    
    if (output_y >= output_height || output_x >= output_width) return;

    const int channels_in = weight_height * weight_width;
    const int channels_out = weight_width;

    // For each output channel
    for (int out_c = 0; out_c < channels_out; out_c++) {
        float sum = 0.0f;
        
        // Compute convolution for this output position
        for (int kh = 0; kh < weight_height; kh++) {
            for (int kw = 0; kw < weight_width; kw++) {
                int input_y = output_y * stride - padding + kh * dilation;
                int input_x = output_x * stride - padding + kw * dilation;
                
                if (input_y >= 0 && input_y < x_height && input_x >= 0 && input_x < x_width) {
                    int input_idx = batch_size * (x_height * x_width) + input_y * x_width + input_x;
                    int weight_idx = out_c * (weight_height * weight_width) + kh * weight_width + kw;
                    
                    sum += x[input_idx] * weight[weight_idx];
                }
            }
        }
        
        // Add bias if present
        if (bias != nullptr) {
            sum += bias[out_c];
        }
        
        // Write output
        int output_idx = batch_size * (output_height * output_width * channels_out) + 
                        out_c * (output_height * output_width) +
                        output_y * output_width + 
                        output_x;
        output[output_idx] = sum;
    }
}

void launch_conv2d_kernel(float* x, float* weight, float* bias, float* output, int stride, int padding, int dilation, int groups, int x_height, int x_width, int weight_height, int weight_width, hipStream_t stream) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((x_width + threadsPerBlock.x - 1) / threadsPerBlock.x, (x_height + threadsPerBlock.y - 1) / threadsPerBlock.y);
    conv2d_kernel<<<numBlocks, threadsPerBlock, 0, stream>>>(x, weight, bias, output, stride, padding, dilation, groups, x_height, x_width, weight_height, weight_width);
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {
    
    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    auto output = torch::empty({x.size(0), weight.size(0), (x.size(2) - weight.size(2) + 2 * padding) / stride + 1, (x.size(3) - weight.size(3) + 2 * padding) / stride + 1}, x.options());

    hipStream_t stream;
    hipStreamCreate(&stream);

    float* x_ptr = x.data_ptr<float>();
    float* weight_ptr = weight.data_ptr<float>();
    float* bias_ptr = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float* output_ptr = output.data_ptr<float>();

    launch_conv2d_kernel(x_ptr, weight_ptr, bias_ptr, output_ptr, stride, padding, dilation, groups, x.size(2), x.size(3), weight.size(2), weight.size(3), stream);

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA forward function for 2D convolution with streams");
}