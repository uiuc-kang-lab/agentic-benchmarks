#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

__inline__ __device__ float warpReduceSum(float val) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__global__ void hybrid_cosine_similarity_loss_kernel(const float* __restrict__ predictions,
                                                      const float* __restrict__ targets,
                                                      float* output,
                                                      int N,
                                                      int D) {
    extern __shared__ float s_data[];
    int row = blockIdx.x;
    int tid = threadIdx.x;
    int stride = blockDim.x;

    const float* pred_row = predictions + row * D;
    const float* target_row = targets + row * D;

    float sum_dot = 0.0f;
    float sum_pred_sq = 0.0f;
    float sum_target_sq = 0.0f;

    for (int i = tid; i < D; i += stride) {
        float p = pred_row[i];
        float t = target_row[i];
        sum_dot += p * t;
        sum_pred_sq += p * p;
        sum_target_sq += t * t;
    }

    sum_dot = warpReduceSum(sum_dot);
    sum_pred_sq = warpReduceSum(sum_pred_sq);
    sum_target_sq = warpReduceSum(sum_target_sq);

    int lane = tid & (warpSize - 1);
    int warp_id = tid >> 5;

    if (lane == 0) {
        s_data[warp_id] = sum_dot;
        s_data[warp_id + 32] = sum_pred_sq;
        s_data[warp_id + 64] = sum_target_sq;
    }

    __syncthreads();

    if (tid < warpSize) {
        int num_warps = (blockDim.x + warpSize - 1) / warpSize;
        float final_dot = (tid < num_warps) ? s_data[tid] : 0.0f;
        float final_pred_sq = (tid < num_warps) ? s_data[tid + 32] : 0.0f;
        float final_target_sq = (tid < num_warps) ? s_data[tid + 64] : 0.0f;

        final_dot = warpReduceSum(final_dot);
        final_pred_sq = warpReduceSum(final_pred_sq);
        final_target_sq = warpReduceSum(final_target_sq);

        if (tid == 0) {
            const float eps = 1e-8f;
            float norm_pred = sqrtf(final_pred_sq);
            float norm_target = sqrtf(final_target_sq);
            float denominator = norm_pred * norm_target;
            denominator = fmaxf(denominator, eps);
            float cos_sim = final_dot / denominator;
            atomicAdd(output, (1.0f - cos_sim) / N);
        }
    }
}

torch::Tensor hybrid_cosine_similarity_loss_forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.dim() == 2, "predictions must be 2D");
    TORCH_CHECK(targets.dim() == 2, "targets must be 2D");
    TORCH_CHECK(predictions.sizes() == targets.sizes(), "Input tensors must have the same shape");
    TORCH_CHECK(predictions.scalar_type() == torch::kFloat32, "predictions must be float32");
    TORCH_CHECK(targets.scalar_type() == torch::kFloat32, "targets must be float32");

    int N = predictions.size(0);
    int D = predictions.size(1);

    auto output = torch::zeros({1}, predictions.options());
    const int block_size = 256;
    size_t shared_mem = 96 * sizeof(float); // 3 * 32 floats

    hybrid_cosine_similarity_loss_kernel<<<N, block_size, shared_mem>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        N,
        D
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &hybrid_cosine_similarity_loss_forward, "Hybrid Cosine Similarity Loss Forward (CUDA)");
}