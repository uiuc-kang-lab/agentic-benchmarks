#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>

__global__ void conv_transpose2d_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    float* __restrict__ output,
    int x_height, int x_width,
    int weight_height, int weight_width,
    int out_height, int out_width,
    int stride_h, int stride_w,
    int padding_h, int padding_w,
    int dilation_h, int dilation_w,
    int groups) {

    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (out_x < out_width && out_y < out_height) {
        float value = 0.0f;
        for (int g = 0; g < groups; ++g) {
            for (int k_h = 0; k_h < weight_height; ++k_h) {
                for (int k_w = 0; k_w < weight_width; ++k_w) {
                    int in_x = (out_x - padding_w + k_w * dilation_w) / stride_w;
                    int in_y = out_y - k_h * dilation_h + padding_h;
                    if (in_x % stride_w == 0 && in_y % stride_h == 0) {
                        in_x /= stride_w;
                        in_y /= stride_h;
                        if (in_x >= 0 && in_x < x_width && in_y >= 0 && in_y < x_height) {
                            int x_index = ((g * x_height + in_y) * x_width + in_x);
                            int weight_index = ((g * weight_height + k_h) * weight_width + k_w);
                            value += x[x_index] * weight[weight_index];
                        }
                    }
                }
            }
        }
        atomicAdd(&output[out_y * out_width + out_x], value);
    }
}

void conv_transpose2d_cuda(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor output,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> dilation,
    int64_t groups) {

    const int x_height = x.size(2);
    const int x_width = x.size(3);
    const int weight_height = weight.size(2);
    const int weight_width = weight.size(3);
    const int out_height = output.size(2);
    const int out_width = output.size(3);

    const dim3 threads(16, 16);
    const dim3 blocks((out_width + threads.x - 1) / threads.x, (out_height + threads.y - 1) / threads.y);

    conv_transpose2d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        output.data_ptr<float>(),
        x_height, x_width,
        weight_height, weight_width,
        out_height, out_width,
        stride[0], stride[1],
        padding[0], padding[1],
        dilation[0], dilation[1],
        groups);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_cuda, "ConvTranspose2D forward (CUDA)");
}