#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32

// Device function to load a tile from matrix A into shared memory
__device__ inline void loadTileA(const float* A, float tileA[TILE_SIZE][TILE_SIZE], int M, int K, int blockRow, int tileIdx) {
    int row = blockRow + threadIdx.y;
    int col = tileIdx * TILE_SIZE + threadIdx.x;
    if(row < M && col < K) {
        tileA[threadIdx.y][threadIdx.x] = A[row * K + col];
    } else {
        tileA[threadIdx.y][threadIdx.x] = 0.0f;
    }
}

// Device function to load a tile from matrix B into shared memory
__device__ inline void loadTileB(const float* B, float tileB[TILE_SIZE][TILE_SIZE], int K, int N, int blockCol, int tileIdx) {
    int row = tileIdx * TILE_SIZE + threadIdx.y;
    int col = blockCol + threadIdx.x;
    if(row < K && col < N) {
        tileB[threadIdx.y][threadIdx.x] = B[row * N + col];
    } else {
        tileB[threadIdx.y][threadIdx.x] = 0.0f;
    }
}

// Kernel function performing tiled matrix multiplication using modular device functions
__global__ void matmul_kernel(const float* __restrict__ A,
                                const float* __restrict__ B,
                                float* __restrict__ C,
                                int M, int K, int N) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];
    
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float value = 0.0f;
    
    int numTiles = (K + TILE_SIZE - 1) / TILE_SIZE;
    int blockRow = blockIdx.y * TILE_SIZE;
    int blockCol = blockIdx.x * TILE_SIZE;
    
    for (int t = 0; t < numTiles; t++) {
        loadTileA(A, tileA, M, K, blockRow, t);
        loadTileB(B, tileB, K, N, blockCol, t);
        __syncthreads();
        
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            value += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }
        __syncthreads();
    }
    
    if (row < M && col < N) {
        C[row * N + col] = value;
    }
}

// The forward function wraps the kernel launch
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "Tensor A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Tensor B must be a CUDA tensor");
    TORCH_CHECK(A.is_contiguous(), "Tensor A must be contiguous");
    TORCH_CHECK(B.is_contiguous(), "Tensor B must be contiguous");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    auto C = torch::zeros({M, N}, A.options());

    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);

    matmul_kernel<<<grid, block>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        M, K, N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Modular tiled matrix multiplication (CUDA)");
}
