#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Device inline function to compute GELU activation.
__device__ inline float gelu_activation(float x) {
    const float sqrt_2_over_pi = 0.7978845608f; // sqrt(2/pi)
    const float coeff = 0.044715f;
    float x_cubed = x * x * x;
    float inner = (x + coeff * x_cubed) * sqrt_2_over_pi;
    return 0.5f * x * (1.0f + tanhf(inner));
}

// Kernel to process input in float4 vectorized chunks
__global__ void gelu_kernel_vector(const float4* __restrict__ x, float4* __restrict__ y, int vec_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < vec_size) {
        float4 v = __ldg(&x[i]);
        v.x = gelu_activation(v.x);
        v.y = gelu_activation(v.y);
        v.z = gelu_activation(v.z);
        v.w = gelu_activation(v.w);
        y[i] = v;
    }
}

// Fallback scalar kernel for remaining elements
__global__ void gelu_kernel_scalar(const float* __restrict__ x, float* __restrict__ y, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = gelu_activation(x[i]);
    }
}

// Forward function accessible from Python
torch::Tensor gelu_forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(x.is_contiguous(), "Input tensor must be contiguous");

    auto y = torch::empty_like(x);
    int n = x.numel();

    // Process most of the tensor with vectorized float4 loads/stores
    int vec_size = n / 4;  // number of float4 vectors
    int remainder = n % 4;

    const int threads = 256;
    if(vec_size > 0) {
        int blocks = (vec_size + threads - 1) / threads;
        const float4* x_vec = reinterpret_cast<const float4*>(x.data_ptr<float>());
        float4* y_vec = reinterpret_cast<float4*>(y.data_ptr<float>());
        gelu_kernel_vector<<<blocks, threads>>>(x_vec, y_vec, vec_size);
    }

    // Process any remaining elements with the scalar kernel
    if(remainder > 0) {
        int offset = vec_size * 4;
        int blocks_rem = (remainder + threads - 1) / threads;
        gelu_kernel_scalar<<<blocks_rem, threads>>>(x.data_ptr<float>() + offset, y.data_ptr<float>() + offset, remainder);
    }

    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &gelu_forward, "Optimized GELU combined CUDA implementation");
}
