#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Store frequently accessed constants in constant memory
__constant__ float kThreshold = 1.0f;
__constant__ float kHalf = 0.5f;

// Kernel using vectorized loads and constant memory for frequently used parameters
__global__ void smooth_l1_loss_const_kernel(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    int n_elements
) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;
    int stride = gridDim.x * blockDim.x;
    float thread_sum = 0.0f;

    // Process data in groups of 4 using vectorized loads
    int vec_count = n_elements / 4;
    const float4* pred4 = reinterpret_cast<const float4*>(predictions);
    const float4* targ4 = reinterpret_cast<const float4*>(targets);

    for (int i = idx; i < vec_count; i += stride) {
        float4 p = __ldg(pred4 + i);
        float4 t = __ldg(targ4 + i);

        float diff = p.x - t.x;
        float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < kThreshold) ? kHalf * diff * diff : abs_diff - kHalf;

        diff = p.y - t.y;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < kThreshold) ? kHalf * diff * diff : abs_diff - kHalf;

        diff = p.z - t.z;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < kThreshold) ? kHalf * diff * diff : abs_diff - kHalf;

        diff = p.w - t.w;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < kThreshold) ? kHalf * diff * diff : abs_diff - kHalf;
    }

    // Process remaining scalar elements
    int remainder_start = vec_count * 4;
    for (int i = remainder_start + idx; i < n_elements; i += stride) {
        float diff = __ldg(predictions + i) - __ldg(targets + i);
        float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < kThreshold) ? kHalf * diff * diff : abs_diff - kHalf;
    }

    // Block-level reduction using shared memory
    __shared__ float shared_mem[256];
    shared_mem[tid] = thread_sum;
    __syncthreads();
    
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_mem[tid] += shared_mem[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        // Divide total loss by the number of elements and atomically add to output
        atomicAdd(output, shared_mem[0] / n_elements);
    }
}

// Host function wrapper
torch::Tensor smooth_l1_loss_cuda_const(
    torch::Tensor predictions,
    torch::Tensor targets
) {
    TORCH_CHECK(predictions.sizes() == targets.sizes(), "Input tensors must have the same shape");
    TORCH_CHECK(predictions.is_contiguous() && targets.is_contiguous(), "Input tensors must be contiguous");
    TORCH_CHECK(predictions.device().is_cuda() && targets.device().is_cuda(), "Inputs must be CUDA tensors");

    int n_elements = predictions.numel();
    auto output = torch::zeros({1}, predictions.options());

    const int block_size = 256;
    int grid_size = (n_elements / 4 + block_size - 1) / block_size;
    grid_size = grid_size > 0 ? grid_size : 1;

    smooth_l1_loss_const_kernel<<<grid_size, block_size>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n_elements
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &smooth_l1_loss_cuda_const, "Smooth L1 Loss (CUDA) with constant memory for parameters");
}
