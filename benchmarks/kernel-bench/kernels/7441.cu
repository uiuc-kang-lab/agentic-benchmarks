#include "hip/hip_runtime.h"
#include <torch/extension.h>

// Kernel with stride loop optimization for bias addition
__global__ void add_bias_kernel_stride(float* output, const float* bias, int total, int C_out, int H_out, int W_out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < total; i += stride) {
        int oc = (i / (H_out * W_out)) % C_out;
        output[i] += bias[oc];
    }
}

// Forward function definition
torch::Tensor conv_transpose2d_forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int64_t stride,
    int64_t padding,
    int64_t output_padding,
    int64_t groups) {

    // Ensure inputs are on CUDA and contiguous
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA");
    TORCH_CHECK(x.is_contiguous(), "Input tensor must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "Weight tensor must be contiguous");

    if (bias.has_value()) {
        TORCH_CHECK(bias.value().is_cuda(), "Bias tensor must be on CUDA");
        TORCH_CHECK(bias.value().is_contiguous(), "Bias tensor must be contiguous");
    }

    // Perform the main conv_transpose2d computation using the built-in function
    auto output = at::conv_transpose2d(
        x,
        weight,
        bias,
        {stride, stride},                   // stride
        {padding, padding},                 // padding
        {output_padding, output_padding},   // output_padding
        groups
    );

    // If bias is provided, execute the bias addition kernel with stride loop optimization
    if (bias.has_value()) {
        int N = x.size(0);
        int C_out = weight.size(1);
        int H_out = output.size(2);
        int W_out = output.size(3);
        int total_output = N * C_out * H_out * W_out;

        // Define block and grid size
        int block_size = 256;
        int grid_size = (total_output + block_size - 1) / block_size;

        add_bias_kernel_stride<<<grid_size, block_size>>>(
            output.data_ptr<float>(),
            bias.value().data_ptr<float>(),
            total_output, C_out, H_out, W_out
        );
        hipDeviceSynchronize();
    }

    return output;
}

// Pybind11 module definition
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward, "ConvTranspose2d forward (CUDA) - stride loop optimized");
}
