#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized kernel with block size experimentation for better performance
__global__ void optimized_block_size_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int N
) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= N || col >= N) return;

    if (row < col) {
        C[row * N + col] = 0.0f;
    } else {
        float sum = 0.0f;
        for (int k = col; k <= row; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    const int N = A.size(0);
    auto C = torch::empty_like(A);

    // Experiment with block size 64x16 for optimal performance
    const int bx = 64, by = 16;
    dim3 threads(bx, by);
    dim3 blocks((N + bx - 1) / bx, (N + by - 1) / by);

    optimized_block_size_kernel<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized block size triangular matmul (CUDA)");
}
