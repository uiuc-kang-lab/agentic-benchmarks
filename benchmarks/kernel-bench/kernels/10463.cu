#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__device__ __forceinline__ float warp_scan(float val, const unsigned mask = 0xffffffff) {
    #pragma unroll
    for (int offset = 1; offset < 32; offset *= 2) {
        float n = __shfl_up_sync(mask, val, offset);
        if (threadIdx.x % 32 >= offset) val += n;
    }
    return val;
}

__global__ void cumsum_kernel_optimized(const float* __restrict__ input,
                            float* __restrict__ output,
                            float* __restrict__ warp_sums,
                            const int inner_size,
                            const int stride) {
    const int idx = blockIdx.x;
    const int outer_idx = idx / inner_size;
    const int inner_idx = idx % inner_size;
    const int lane_id = threadIdx.x % 32;
    const int warp_id = threadIdx.x / 32;
    const int warps_per_block = blockDim.x / 32;
    
    // Base index for this thread's work
    const int base_idx = outer_idx * stride * inner_size + inner_idx;
    
    // Process elements in chunks of 32 (warp size)
    for (int warp_start = warp_id * 32; warp_start < stride; warp_start += warps_per_block * 32) {
        float val = 0.0f;
        const int pos = warp_start + lane_id;
        
        if (pos < stride) {
            val = input[base_idx + pos * inner_size];
        }
        
        // Perform warp-level scan
        val = warp_scan(val);
        
        // Last thread in warp stores sum for next warp
        if (lane_id == 31 && pos < stride) {
            warp_sums[outer_idx * ((stride + 31)/32) + warp_start/32] = val;
        }
        
        // Synchronize only when necessary
        if (warp_start + 32 < stride) {
            __syncthreads();
        }
        
        // Add previous warps' sums
        if (pos < stride && warp_start > 0) {
            float prev_sum = 0.0f;
            #pragma unroll 4
            for (int w = 0; w < warp_start / 32; w++) {
                prev_sum += warp_sums[outer_idx * ((stride + 31)/32) + w];
            }
            val += prev_sum;
        }
        
        // Store result
        if (pos < stride) {
            output[base_idx + pos * inner_size] = val;
        }
    }
}

torch::Tensor forward(torch::Tensor x, int dim) {
    CHECK_INPUT(x);
    
    auto output = torch::empty_like(x);
    int ndim = x.dim();
    dim = (dim + ndim) % ndim;
    
    int outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= x.size(i);
    }
    
    int inner_size = 1;
    for (int i = dim + 1; i < ndim; i++) {
        inner_size *= x.size(i);
    }
    
    int stride = x.size(dim);
    
    // Allocate temporary storage for warp sums
    auto warp_sums = torch::empty({outer_size * ((stride + 31)/32)}, x.options());
    
    const int total_blocks = outer_size * inner_size;
    const int threads_per_block = 256; // Use 8 warps per block
    
    cumsum_kernel_optimized<<<total_blocks, threads_per_block>>>(
        x.data_ptr<float>(),
        output.data_ptr<float>(),
        warp_sums.data_ptr<float>(),
        inner_size,
        stride
    );
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized CUDA cumulative sum with minimal synchronization");
}