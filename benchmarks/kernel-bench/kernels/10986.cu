#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static const int BLOCK_SIZE = 256;

template <typename scalar_t>
__global__ void mse_forward_kernel_min_sync(
    const scalar_t* __restrict__ preds,
    const scalar_t* __restrict__ tgts,
    double* __restrict__ sum_out,
    const int64_t num_elements
) {
    // Each thread maintains its own accumulator
    double thread_sum = 0.0;
    
    // Process multiple elements per thread using grid-stride loop
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; 
         idx < num_elements; 
         idx += blockDim.x * gridDim.x) {
        double diff = static_cast<double>(preds[idx]) - static_cast<double>(tgts[idx]);
        thread_sum += diff * diff;
    }

    // Warp-level reduction first (no sync needed within a warp)
    unsigned int mask = 0xffffffff;
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2) {
        thread_sum += __shfl_down_sync(mask, thread_sum, offset);
    }

    // Only the first thread in each warp writes to shared memory
    __shared__ double warp_sums[8];  // For 256 threads = 8 warps
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;
    
    if (lane_id == 0) {
        warp_sums[warp_id] = thread_sum;
    }
    
    // Single sync point needed here for shared memory consistency
    __syncthreads();

    // Final reduction by first warp only
    if (threadIdx.x < 8) {
        double sum = warp_sums[threadIdx.x];
        
        // Warp-level reduction of final sums (no sync needed)
        #pragma unroll
        for (int offset = 4; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xff, sum, offset);
        }

        if (threadIdx.x == 0) {
            atomicAdd(sum_out, sum);
        }
    }
}

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.is_cuda(), "predictions must be a CUDA tensor");
    TORCH_CHECK(targets.is_cuda(), "targets must be a CUDA tensor");
    TORCH_CHECK(predictions.numel() == targets.numel(),
                "predictions and targets must have the same number of elements");

    const int64_t num_elements = predictions.numel();
    auto accumulator = torch::zeros({1}, predictions.options().dtype(at::kDouble));

    const int grid_size = std::min(1024, (int)((num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE));

    AT_DISPATCH_FLOATING_TYPES(predictions.scalar_type(), "mse_forward_cuda", ([&] {
        mse_forward_kernel_min_sync<scalar_t><<<grid_size, BLOCK_SIZE>>>(
            predictions.data_ptr<scalar_t>(),
            targets.data_ptr<scalar_t>(),
            accumulator.data_ptr<double>(),
            num_elements
        );
    }));

    auto result = accumulator.div_(static_cast<double>(num_elements));
    return result.to(predictions.dtype());
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "MSE forward (CUDA) with minimal synchronization");
}