#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void max_pool2d_kernel_warp_divergence(
    const scalar_t* input,
    scalar_t* output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    const int ow = output_idx % output_width;
    const int oh = (output_idx / output_width) % output_height;
    const int c = (output_idx / (output_width * output_height)) % channels;
    const int b = output_idx / (output_width * output_height * channels);

    int ih_start = oh * stride - padding;
    int iw_start = ow * stride - padding;
    int ih_end = min(ih_start + kernel_size * dilation, input_height);
    int iw_end = min(iw_start + kernel_size * dilation, input_width);
    ih_start = max(ih_start, 0);
    iw_start = max(iw_start, 0);

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    for (int ih = ih_start; ih < ih_end; ih += dilation) {
        for (int iw = iw_start; iw < iw_end; iw += dilation) {
            const int input_idx = b * (channels * input_height * input_width) +
                                c * (input_height * input_width) +
                                ih * input_width +
                                iw;
            max_val = max(max_val, input[input_idx]);
        }
    }

    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward_warp_divergence(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward_warp_divergence", ([&] {
        max_pool2d_kernel_warp_divergence<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward_warp_divergence, "Max Pool 2D forward with minimized warp divergence (CUDA)");
}