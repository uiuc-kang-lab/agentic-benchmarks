#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// This kernel computes the upper triangular matrix multiplication (C = A * B)
// where only elements with row <= col are evaluated. It uses a balanced workload
// distribution by assigning each thread to compute multiple elements in a row-major
// order, ensuring that all threads are utilized effectively.

__global__ void balanced_workload_upper_triangular_kernel(const float* __restrict__ A,
                                                           const float* __restrict__ B,
                                                           float* __restrict__ C,
                                                           int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col_start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int col = col_start; col < N; col += stride) {
        if (row < N && row <= col) {
            float sum = 0.0f;
            // Loop from k = row to k = col in order to follow the upper triangular property
            for (int k = row; k <= col; ++k) {
                float a_val = __ldg(&A[row * N + k]);
                float b_val = __ldg(&B[k * N + col]);
                sum += a_val * b_val;
            }
            C[row * N + col] = sum;
        }
    }
}

// Host function, exposed via pybind11, that wraps the kernel invocation
// It creates a zero tensor for C, launches the kernel, and returns C.

torch::Tensor balanced_workload_upper_triangular_matmul(torch::Tensor A, torch::Tensor B) {
    int N = A.size(0);
    auto C = torch::zeros_like(A);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    balanced_workload_upper_triangular_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &balanced_workload_upper_triangular_matmul, "Balanced workload upper triangular matrix multiplication");
}
