#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define NUM_STREAMS 4

__global__ void leaky_relu_kernel(const float* x, float* out, float negative_slope, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = x[idx] > 0 ? x[idx] : x[idx] * negative_slope;
    }
}

torch::Tensor leaky_relu_forward(torch::Tensor x, float negative_slope) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int n = x.numel();
    int chunk_size = (n + NUM_STREAMS - 1) / NUM_STREAMS;

    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    const int threads = 256;
    
    for (int i = 0; i < NUM_STREAMS; i++) {
        int offset = i * chunk_size;
        int current_size = min(chunk_size, n - offset);
        if (current_size <= 0) break;
        
        const int blocks = (current_size + threads - 1) / threads;
        
        leaky_relu_kernel<<<blocks, threads, 0, streams[i]>>>(
            x.data_ptr<float>() + offset,
            out.data_ptr<float>() + offset,
            negative_slope,
            current_size
        );
    }

    // Synchronize all streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &leaky_relu_forward, "LeakyReLU forward (CUDA)");
}