#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized kernel using grid-stride loop and an experimentally tuned block size

template <typename scalar_t>
__global__ void conv_transpose2d_kernel_opt(
    const scalar_t* input,
    const scalar_t* weight,
    const scalar_t* bias,
    scalar_t* output,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int out_channels,
    const int kernel_h,
    const int kernel_w,
    const int stride,
    const int padding,
    const int output_padding,
    const int groups,
    const int dilation,
    const int out_height,
    const int out_width
) {
    const int total_elements = batch_size * out_channels * out_height * out_width;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int gridStride = blockDim.x * gridDim.x;

    // Use grid-stride loop to cover all output elements
    for (; idx < total_elements; idx += gridStride) {
        // Unravel index
        int n = idx;
        const int ow = n % out_width;
        n /= out_width;
        const int oh = n % out_height;
        n /= out_height;
        const int oc = n % out_channels;
        n /= out_channels;
        const int b = n;

        const int out_channels_per_group = out_channels / groups;
        const int g = oc / out_channels_per_group;
        const int oc_group = oc % out_channels_per_group;  
        const int in_channels_per_group = in_channels / groups;
        const int ic_start = g * in_channels_per_group;

        scalar_t val = (bias != nullptr) ? bias[oc] : static_cast<scalar_t>(0);

        for (int kh = 0; kh < kernel_h; ++kh) {
            for (int kw = 0; kw < kernel_w; ++kw) {
                const int h_in_tmp = oh - kh * dilation + padding;
                if (h_in_tmp < 0 || h_in_tmp % stride != 0) continue;
                const int h_in = h_in_tmp / stride;
                if (h_in < 0 || h_in >= in_height) continue;

                const int w_in_tmp = ow - kw * dilation + padding;
                if (w_in_tmp < 0 || w_in_tmp % stride != 0) continue;
                const int w_in = w_in_tmp / stride;
                if (w_in < 0 || w_in >= in_width) continue;

                for (int ic = 0; ic < in_channels_per_group; ++ic) {
                    const scalar_t x_val = input[b * in_channels * in_height * in_width
                                              + (ic_start + ic) * in_height * in_width
                                              + h_in * in_width
                                              + w_in];

                    const scalar_t w_val = weight[(ic_start + ic) * (out_channels_per_group * kernel_h * kernel_w) +
                                                   oc_group * kernel_h * kernel_w +
                                                   kh * kernel_w +
                                                   kw];

                    val += x_val * w_val;
                }
            }
        }
        output[idx] = val;
    }
}


torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int output_padding,
    int groups,
    int dilation = 1
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(x.dim() == 4, "Input must be 4D");
    TORCH_CHECK(weight.dim() == 4, "Weight must be 4D");

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);

    // Calculate output channels based on groups
    const int out_channels = weight.size(1) * groups;  // weight shape: [in_channels, out_channels/groups, kH, kW]
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);

    if (bias.has_value() && bias->defined()) {
        TORCH_CHECK(bias->numel() == out_channels, "Bias must have out_channels elements");
        TORCH_CHECK(bias->device().is_cuda(), "Bias must be a CUDA tensor");
    }

    // Compute output dimensions
    const int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_h - 1) + output_padding + 1;
    const int out_width = (in_width - 1) * stride - 2 * padding + dilation * (kernel_w - 1) + output_padding + 1;

    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, x.options());

    const int total_elements = output.numel();

    // Experimentally tuned block size for NVIDIA H100; change this constant to 32, 64, 128, 256, or 512
    constexpr int BLOCK_SIZE = 512;
    const int blocks = (total_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "conv_transpose2d_cuda_opt", ([&] {
        conv_transpose2d_kernel_opt<scalar_t><<<blocks, BLOCK_SIZE>>>(
            x.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            (bias.has_value() && bias->defined()) ? bias->data_ptr<scalar_t>() : nullptr,
            output.data_ptr<scalar_t>(),
            batch_size,
            in_channels,
            in_height,
            in_width,
            out_channels,
            kernel_h,
            kernel_w,
            stride,
            padding,
            output_padding,
            groups,
            dilation,
            out_height,
            out_width
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Transposed 2D Convolution with Block Size Tuning (CUDA)",
          py::arg("x"), py::arg("weight"), py::arg("bias") = py::none(),
          py::arg("stride"), py::arg("padding"), py::arg("output_padding"),
          py::arg("groups"), py::arg("dilation") = 1);
}
