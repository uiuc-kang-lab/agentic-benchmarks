#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void sigmoid_kernel(const scalar_t* __restrict__ input,
                               scalar_t* __restrict__ output,
                               const int64_t size) {
    // Align thread block size to warp size for coalesced memory access
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int stride = blockDim.x * gridDim.x;
    
    // Process multiple elements per thread with coalesced access pattern
    for(int i = tid; i < size; i += stride) {
        float val = static_cast<float>(-input[i]);
        float exp_val = expf(val);
        float r = 1.0f / (1.0f + exp_val);
        output[i] = static_cast<scalar_t>(r);
    }
}

torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int64_t size = input.numel();

    // Use warp size (32) aligned thread blocks for better memory coalescing
    const int threads = 256;  // Multiple of warp size (32)
    const int blocks = std::min(65535, (int)((size + threads - 1) / threads));

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sigmoid_kernel", [&] {
        const auto* input_data = input.data_ptr<scalar_t>();
        auto* output_data = output.data_ptr<scalar_t>();

        sigmoid_kernel<scalar_t><<<blocks, threads>>>(input_data, output_data, size);
    });

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Sigmoid forward (CUDA)");
}