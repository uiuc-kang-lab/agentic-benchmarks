#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

__constant__ int const_params[4];

template <typename scalar_t>
__device__ __forceinline__ int compute_input_index(
    int b, int c, int h, int w,
    int channels, int height, int width
) {
    return b * (channels * height * width) +
           c * (height * width) +
           h * width +
           w;
}

template <typename scalar_t>
__device__ __forceinline__ bool is_valid_input_location(
    int h, int w,
    int input_height, int input_width
) {
    return (h >= 0 && h < input_height && w >= 0 && w < input_width);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_max_pool(
    const scalar_t* __restrict__ input,
    int oh, int ow,
    int b, int c,
    int input_height, int input_width,
    int channels
) {
    const int kernel_size = const_params[0];
    const int stride = const_params[1];
    const int padding = const_params[2];
    const int dilation = const_params[3];

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    #pragma unroll
    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = oh * stride - padding + kh * dilation;
        
        #pragma unroll
        for (int kw = 0; kw < kernel_size; kw++) {
            const int iw = ow * stride - padding + kw * dilation;

            if (is_valid_input_location<scalar_t>(ih, iw, input_height, input_width)) {
                const int input_idx = compute_input_index<scalar_t>(
                    b, c, ih, iw,
                    channels, input_height, input_width
                );
                max_val = max(max_val, __ldg(&input[input_idx]));
            }
        }
    }

    return max_val;
}

template <typename scalar_t>
__global__ void optimized_max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size * channels * output_height * output_width) return;

    const int ow = idx % output_width;
    const int oh = (idx / output_width) % output_height;
    const int c = (idx / (output_width * output_height)) % channels;
    const int b = idx / (output_width * output_height * channels);

    const scalar_t max_val = compute_max_pool<scalar_t>(
        input, oh, ow, b, c,
        input_height, input_width, channels
    );

    output[idx] = max_val;
}

torch::Tensor optimized_max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int params[4] = {kernel_size, stride, padding, dilation};
    hipMemcpyToSymbol(HIP_SYMBOL(const_params), params, sizeof(int) * 4);

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "optimized_max_pool2d_cuda_forward", ([&] {
        optimized_max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &optimized_max_pool2d_cuda_forward, "Optimized Max Pool 2D forward (CUDA)");
}