#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <iostream>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void matmul_warp_level(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < N) {  // Boundary check
        float sum = 0.0f;
        for (int i = 0; i < K; ++i) {
            float a = A[row * K + i];
            float b = B[i * N + col];
            sum += a * b;
        }

        // Each thread computes its own complete dot product
        C[row * N + col] = sum;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((N + 31) / 32, (M + 31) / 32);

    matmul_warp_level<<<blocksPerGrid, threadsPerBlock>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, N, K);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication with warp-level primitives (CUDA)");
}