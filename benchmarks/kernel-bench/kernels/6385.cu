#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel that uses warp-level primitives to perform reduction over the specified dimension.

template <typename scalar_t>
__global__ void warp_sum_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t reduce_size,
    int64_t inner_size,
    int64_t total_output) {

    // Each block handles one output element (one (outer, inner) pair).
    int idx = blockIdx.x;  // index for output element
    if (idx >= total_output) return;

    // Determine corresponding outer and inner indices
    int outer_idx = idx / inner_size;
    int inner_idx = idx % inner_size;

    scalar_t sum = 0;
    // Use lane id of the warp; assume blockDim.x == warpSize (32 threads)
    int lane = threadIdx.x;

    // Pre-calculate base offset to avoid redundant computations
    const int64_t base_offset = outer_idx * reduce_size * inner_size + inner_idx;
    
    // Each thread in the warp sums elements from the reduction dim in a strided manner
    for (int i = lane; i < reduce_size; i += warpSize) {
        int64_t offset = base_offset + i * inner_size;
        // Use __ldg() for read-only access to global memory
        sum += __ldg(&input[offset]);
    }

    // Use warp-level shuffle to reduce the partial sums within the warp
    // Cache the current partial sum in register
    scalar_t partial = sum;
    const unsigned int mask = 0xffffffff;
    
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        const scalar_t n = __shfl_down_sync(mask, partial, offset);
        partial += n;
    }
    sum = partial;

    // The first lane writes the result
    if (lane == 0) {
        // Ensure memory access is aligned
        atomicAdd(&output[idx], sum);
    }
}

// Host function wrapping the kernel launch

torch::Tensor sum_reduce_cuda(torch::Tensor input, int64_t dim) {
    // Adjust negative dimensions
    if (dim < 0) dim += input.dim();
    auto sizes = input.sizes().vec();
    int64_t reduce_size = sizes[dim];

    // Compute outer_size: product of dimensions before the reduction dim
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }

    // Compute inner_size: product of dimensions after the reduction dim
    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }

    // Prepare output tensor with reduce dimension set to 1
    sizes[dim] = 1;
    auto output = torch::empty(sizes, input.options());

    // Total number of output elements
    int64_t total_output = outer_size * inner_size;

    // Launch one warp (32 threads) per output element
    const int threads = 32;  // warp size
    const int blocks = total_output;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sum_reduce_cuda", ([&] {
        warp_sum_reduce_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            reduce_size,
            inner_size,
            total_output
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &sum_reduce_cuda, "Sum reduction forward (CUDA) using warp-level primitives");
}
