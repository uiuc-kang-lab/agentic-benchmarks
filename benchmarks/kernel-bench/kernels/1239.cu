#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

__global__ void einsum_kernel_unroll(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int BATCH, int I, int J, int L, int K
) {
    extern __shared__ float shared_B[];

    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_idx >= BATCH * I * J * K) return;

    int k = global_idx % K;
    int remainder = global_idx / K;
    int j = remainder % J;
    remainder /= J;
    int i = remainder % I;
    int b = remainder / I;

    float sum = 0.0f;
    for(int l = 0; l < L; ++l) {
        // Cooperatively load B into shared memory
        for (int idx = threadIdx.x; idx < K; idx += blockDim.x) {
            shared_B[idx] = B[l*K + idx];
        }
        __syncthreads();

        int a_offset = b * I*J*L + i*J*L + j*L + l;
        #pragma unroll
        for (int u = 0; u < 4; ++u) { // unroll factor of 4
            if (l + u < L) {
                sum += A[a_offset + u] * shared_B[k];
            }
        }
        __syncthreads();
    }

    C[global_idx] = sum;
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(A.dim() == 4, "A must be 4D");
    TORCH_CHECK(B.dim() == 2, "B must be 2D");
    TORCH_CHECK(A.size(3) == B.size(0), "Dimension mismatch in l");

    int BATCH = A.size(0), I = A.size(1), J = A.size(2), L = A.size(3);
    int K = B.size(1);
    
    auto C = torch::zeros({BATCH, I, J, K}, A.options());
    int total_elements = BATCH * I * J * K;
    
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;
    
    einsum_kernel_unroll<<<blocks, threads, K * sizeof(float)>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        BATCH, I, J, L, K
    );
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "4D tensor-matrix multiplication with loop unrolling (CUDA)");}
