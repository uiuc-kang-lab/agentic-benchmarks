#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cudnn/Handles.h>
#include <ATen/cudnn/Descriptors.h>
#include <hipDNN.h>

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
#define BLOCK_SIZE_Z 4

// Helper function to map at::ScalarType to hipdnnDataType_t
hipdnnDataType_t getCudnnDataType(at::ScalarType type) {
    switch (type) {
        case at::ScalarType::Float:
            return HIPDNN_DATA_FLOAT;
        case at::ScalarType::Double:
            return HIPDNN_DATA_DOUBLE;
        case at::ScalarType::Half:
            return HIPDNN_DATA_HALF;
        default:
            TORCH_CHECK(false, "Unsupported data type for cuDNN");
    }
}

__global__ void conv3d_kernel_minimal_divergence(
    float* __restrict__ output,
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    int batch_size, int in_channels, int out_channels,
    int in_depth, int in_height, int in_width,
    int kernel_d, int kernel_h, int kernel_w,
    int out_depth, int out_height, int out_width,
    int stride, int padding, int dilation, int groups) {

    int w_out = blockIdx.x * blockDim.x + threadIdx.x;
    int h_out = blockIdx.y * blockDim.y + threadIdx.y;
    int d_out = blockIdx.z * blockDim.z + threadIdx.z;

    if (w_out < out_width && h_out < out_height && d_out < out_depth) {
        for (int c_out = blockIdx.w; c_out < out_channels; c_out += gridDim.w) {
            float sum = 0.0f;

            int group = c_out / (out_channels / groups);
            int in_channels_per_group = in_channels / groups;
            
            for (int ic = 0; ic < in_channels_per_group; ic++) {
                int in_channel = group * in_channels_per_group + ic;

                for (int kd = 0; kd < kernel_d; kd++) {
                    int d_in = d_out * stride - padding + kd * dilation;
                    if (d_in >= 0 && d_in < in_depth) {
                        for (int kh = 0; kh < kernel_h; kh++) {
                            int h_in = h_out * stride - padding + kh * dilation;
                            if (h_in >= 0 && h_in < in_height) {
                                for (int kw = 0; kw < kernel_w; kw++) {
                                    int w_in = w_out * stride - padding + kw * dilation;
                                    if (w_in >= 0 && w_in < in_width) {

                                        int input_index = ((blockIdx.w * in_channels + in_channel) * in_depth + d_in) * in_height * in_width + h_in * in_width + w_in;
                                        int weight_index = (((c_out * in_channels_per_group) + ic) * kernel_d + kd) * kernel_h * kernel_w + kh * kernel_w + kw;

                                        sum += input[input_index] * weight[weight_index];
                                    }
                                }
                            }
                        }
                    }
                }
            }

            if (bias) {
                sum += bias[c_out];
            }

            int output_index = ((blockIdx.w * out_channels + c_out) * out_depth + d_out) * out_height * out_width + h_out * out_width + w_out;
            output[output_index] = sum;
        }
    }
}

at::Tensor forward(
    const at::Tensor& input,
    const at::Tensor& weight,
    const c10::optional<at::Tensor>& bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups
) {
    auto bias = bias_opt.value_or(at::Tensor());
    // Ensure inputs are on CUDA
    TORCH_CHECK(input.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "Weight must be a CUDA tensor");
    TORCH_CHECK(!bias.defined() || bias.is_cuda(), "Bias must be a CUDA tensor");

    // Get input dimensions
    int64_t batch_size = input.size(0);
    int64_t in_channels = input.size(1);
    int64_t in_depth = input.size(2);
    int64_t in_height = input.size(3);
    int64_t in_width = input.size(4);

    // Get weight dimensions
    int64_t out_channels = weight.size(0);
    int64_t kernel_d = weight.size(2);
    int64_t kernel_h = weight.size(3);
    int64_t kernel_w = weight.size(4);

    // Calculate output dimensions
    int64_t out_depth = (in_depth + 2 * padding - dilation * (kernel_d - 1) - 1) / stride + 1;
    int64_t out_height = (in_height + 2 * padding - dilation * (kernel_h - 1) - 1) / stride + 1;
    int64_t out_width = (in_width + 2 * padding - dilation * (kernel_w - 1) - 1) / stride + 1;

    // Prepare output tensor
    auto options = input.options();
    auto output = at::empty({batch_size, out_channels, out_depth, out_height, out_width}, options);

    // Define grid and block dimensions
    dim3 threadsPerBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    dim3 numBlocks(
        (out_width + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X,
        (out_height + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y,
        (out_depth + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z
    );

    conv3d_kernel_minimal_divergence<<<numBlocks, threadsPerBlock>>>(
        output.data_ptr<float>(),
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.defined() ? bias.data_ptr<float>() : nullptr,
        batch_size, in_channels, out_channels,
        in_depth, in_height, in_width,
        kernel_d, kernel_h, kernel_w,
        out_depth, out_height, out_width,
        stride, padding, dilation, groups
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D convolution forward with minimal warp divergence (CUDA)");
}
