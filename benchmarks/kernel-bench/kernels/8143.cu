#include "hip/hip_runtime.h"
/*
Combined Optimized Transposed Convolution 2D Kernel
This kernel integrates the key improvements from two implementations:
1. __restrict__ qualifiers for improved memory access (from Kernel 2).
2. Grid-stride loop with combined conditionals to minimize warp divergence (from Kernel 1).
*/

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


template <typename scalar_t>
__global__ void conv_transpose2d_kernel_combined(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int out_channels,
    const int kernel_h,
    const int kernel_w,
    const int stride,
    const int padding,
    const int output_padding,
    const int groups,
    const int dilation,
    const int out_height,
    const int out_width
) {
    const int total_elements = batch_size * out_channels * out_height * out_width;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int gridStride = blockDim.x * gridDim.x;

    for (; idx < total_elements; idx += gridStride) {
        int n = idx;
        const int ow = n % out_width;
        n /= out_width;
        const int oh = n % out_height;
        n /= out_height;
        const int oc = n % out_channels;
        n /= out_channels;
        const int b = n;

        // Determine group and channel offsets
        const int out_channels_per_group = out_channels / groups;
        const int g = oc / out_channels_per_group;
        const int oc_group = oc % out_channels_per_group;
        const int in_channels_per_group = in_channels / groups;
        const int ic_start = g * in_channels_per_group;

        // Initialize accumulator with bias if provided
        scalar_t accum = (bias != nullptr) ? bias[oc] : static_cast<scalar_t>(0);

        // Loop over the kernel spatial dimensions
        for (int kh = 0; kh < kernel_h; ++kh) {
            int h_in_temp = oh - kh * dilation + padding;
            // Combined condition for h dimension
            if (h_in_temp < 0 || (h_in_temp % stride) != 0) continue;
            int h_in = h_in_temp / stride;
            if (h_in < 0 || h_in >= in_height) continue;

            for (int kw = 0; kw < kernel_w; ++kw) {
                int w_in_temp = ow - kw * dilation + padding;
                if (w_in_temp < 0 || (w_in_temp % stride) != 0) continue;
                int w_in = w_in_temp / stride;
                if (w_in < 0 || w_in >= in_width) continue;

                // Accumulate contributions for each input channel in the current group
                for (int ic = 0; ic < in_channels_per_group; ++ic) {
                    int input_idx = b * (in_channels * in_height * in_width)
                                  + (ic_start + ic) * (in_height * in_width)
                                  + h_in * in_width + w_in;

                    int weight_idx = (ic_start + ic) * (out_channels_per_group * kernel_h * kernel_w)
                                   + oc_group * (kernel_h * kernel_w)
                                   + kh * kernel_w + kw;

                    accum += input[input_idx] * weight[weight_idx];
                }
            } // end kw loop
        } // end kh loop

        output[idx] = accum;
    }
}


// Forward interface

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int output_padding,
    int groups,
    int dilation = 1
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(x.dim() == 4, "Input must be 4D");
    TORCH_CHECK(weight.dim() == 4, "Weight must be 4D");

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);

    // Weight shape: [in_channels, out_channels/groups, kH, kW]
    const int out_channels = weight.size(1) * groups;
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);

    if (bias.has_value() && bias->defined()) {
        TORCH_CHECK(bias->numel() == out_channels, "Bias must have out_channels elements");
        TORCH_CHECK(bias->device().is_cuda(), "Bias must be a CUDA tensor");
    }

    // Compute output dimensions for transposed convolution
    const int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_h - 1) + output_padding + 1;
    const int out_width = (in_width - 1) * stride - 2 * padding + dilation * (kernel_w - 1) + output_padding + 1;

    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, x.options());
    const int total_elements = output.numel();
    
    constexpr int BLOCK_SIZE = 256;
    const int blocks = (total_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "conv_transpose2d_cuda_combined", ([&] {
        conv_transpose2d_kernel_combined<scalar_t><<<blocks, BLOCK_SIZE>>>(
            x.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            (bias.has_value() && bias->defined()) ? bias->data_ptr<scalar_t>() : nullptr,
            output.data_ptr<scalar_t>(),
            batch_size,
            in_channels,
            in_height,
            in_width,
            out_channels,
            kernel_h,
            kernel_w,
            stride,
            padding,
            output_padding,
            groups,
            dilation,
            out_height,
            out_width
        );
    }));

    return output;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Combined Optimized Transposed 2D Convolution (CUDA)",
          py::arg("x"), py::arg("weight"), py::arg("bias") = py::none(),
          py::arg("stride"), py::arg("padding"), py::arg("output_padding"),
          py::arg("groups"), py::arg("dilation") = 1);
}
