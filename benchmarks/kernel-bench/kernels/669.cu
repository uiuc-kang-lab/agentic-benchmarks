#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

// CUDA kernel for matrix multiplication using shared memory tiling with optimized atomic operations
template <typename scalar_t>
__global__ void matmul_cuda_kernel_atomic_optimized(const scalar_t* __restrict__ A, const scalar_t* __restrict__ B,
                                   scalar_t* __restrict__ C, int M, int K, int N) {
    __shared__ scalar_t sA[TILE_WIDTH][TILE_WIDTH];
    __shared__ scalar_t sB[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y; // M dimension
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x; // N dimension

    scalar_t value = 0;

    // Loop over tiles
    for (int t = 0; t < (K + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {
        // Load elements into shared memory
        if (row < M && t * TILE_WIDTH + threadIdx.x < K)
            sA[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_WIDTH + threadIdx.x];
        else
            sA[threadIdx.y][threadIdx.x] = 0;

        if (col < N && t * TILE_WIDTH + threadIdx.y < K)
            sB[threadIdx.y][threadIdx.x] = B[(t * TILE_WIDTH + threadIdx.y) * N + col];
        else
            sB[threadIdx.y][threadIdx.x] = 0;

        __syncthreads(); // Synchronize threads within a block

        for (int i = 0; i < TILE_WIDTH; ++i) {
            value += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads(); // Synchronize threads within a block
    }

    // Use atomic operation only if necessary
    if (row < M && col < N) {
        atomicAdd(&C[row * N + col], value);
    }
}

// Forward function
torch::Tensor module_fn(torch::Tensor A, torch::Tensor B) {
    // Ensure input tensors are CUDA tensors
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input tensor B must be a CUDA tensor");

    // Get matrix dimensions
    int64_t M = A.size(0);
    int64_t K = A.size(1);
    int64_t N = B.size(1);

    // Check dimensions compatibility
    TORCH_CHECK(K == B.size(0), "Inner dimensions of A and B must match");

    // Allocate output tensor
    auto C = torch::zeros({M, N}, A.options()); // Initialize with zeros to use atomicAdd correctly

    // Define block and grid dimensions
    dim3 threads_per_block(TILE_WIDTH, TILE_WIDTH);
    dim3 num_blocks((N + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);

    // Launch the CUDA kernel
    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matmul_cuda_kernel_atomic_optimized", ([&] {
        matmul_cuda_kernel_atomic_optimized<scalar_t><<<num_blocks, threads_per_block>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, K, N);
    }));

    // Wait for all kernels to finish
    hipDeviceSynchronize();

    return C;
}

// Binding code
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &module_fn, "Matrix multiplication forward (CUDA)");
}