#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__inline__ __device__
float warp_reduce_sum(float val) {
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xffffffff, val, offset);
    return val;
}

__global__ void kl_div_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    extern __shared__ float shared_data[];
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    float sum = 0.0f;
    
    // Grid-stride loop
    while (idx < n) {
        float log_pred = log_predictions[idx];
        float target = targets[idx];
        sum += expf(log_pred) - target * log_pred;
        idx += stride;
    }

    // Each thread puts its local sum into shared memory
    shared_data[tid] = sum;
    __syncthreads();

    // Intra-block reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        atomicAdd(output, shared_data[0]);
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    
    // Calculate optimal thread/block configuration
    const int threads_per_block = 256;
    const int max_blocks = 256;
    const int num_blocks = min(max_blocks, (n + threads_per_block - 1) / threads_per_block);
    
    auto output = torch::zeros({1}, log_predictions.options());
    
    kl_div_kernel<<<num_blocks, threads_per_block, threads_per_block * sizeof(float)>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward (CUDA)");
}