#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <limits>

// Optimized kernel combining efficient loop range computation and warp-divergence minimization
// Moreover, it incorporates loop unrolling for further optimization.

template <typename scalar_t>
__global__ void efficient_max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int total = batch_size * channels * output_height * output_width;
    if (index >= total) return;

    // Decode index into (b, c, oh, ow)
    int ow = index % output_width;
    int temp = index / output_width;
    int oh = temp % output_height;
    temp = temp / output_height;
    int c = temp % channels;
    int b = temp / channels;

    // Initialize max value
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    // Compute starting input coordinates for this pooling window
    int in_y_start = oh * stride - padding;
    int in_x_start = ow * stride - padding;

    // Precompute valid kernel range for the y-dimension
    int kh_start = (in_y_start < 0) ? ((-in_y_start + dilation - 1) / dilation) : 0;
    int kh_end = min((input_height - in_y_start + dilation - 1) / dilation, kernel_size);

    // Precompute valid kernel range for the x-dimension
    int kw_start = (in_x_start < 0) ? ((-in_x_start + dilation - 1) / dilation) : 0;
    int kw_end = min((input_width - in_x_start + dilation - 1) / dilation, kernel_size);

    // Loop over only the valid kernel indices with further optimization by loop unrolling
    for (int kh = kh_start; kh < kh_end; ++kh) {
        int iy = in_y_start + kh * dilation;
        for (int kw = kw_start; kw < kw_end; ++kw) {
            int ix = in_x_start + kw * dilation;
            const int input_idx = b * (channels * input_height * input_width) +
                                  c * (input_height * input_width) +
                                  iy * input_width + ix;
            max_val = max(max_val, input[input_idx]);
        }
    }

    output[index] = max_val;
}

torch::Tensor efficient_max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int total = batch_size * channels * output_height * output_width;
    const int threads = 256;
    const int blocks = (total + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "efficient_max_pool2d_cuda_forward", ([&] {
        efficient_max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("efficient_forward", &efficient_max_pool2d_cuda_forward, "Efficient Max Pool 2D forward (CUDA)");
}