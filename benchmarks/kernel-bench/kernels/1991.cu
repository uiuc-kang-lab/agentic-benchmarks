#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void dynamic_block_size_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int N
) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= N || col >= N) return;

    if (row < col) {
        C[row * N + col] = 0.0f;
    } else {
        float sum = 0.0f;
        for (int k = col; k <= row; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

at::Tensor forward_dynamic(at::Tensor A, at::Tensor B, int block_size) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    const int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threads(block_size, block_size / 4);
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);

    dynamic_block_size_kernel<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_dynamic, "Dynamic block size triangular matmul (CUDA)");
}
