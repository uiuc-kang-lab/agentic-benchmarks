#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Kernel function for max pooling with loop unrolling
template <typename scalar_t>
__global__ void max_pool3d_forward_kernel(
    const scalar_t* input,
    scalar_t* output,
    int64_t* indices,
    const int batch_size,
    const int channels,
    const int input_d, const int input_h, const int input_w,
    const int output_d, const int output_h, const int output_w,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation) {
    
    // Calculate thread indices for better memory coalescing
    const int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_threads = gridDim.x * blockDim.x;
    
    // Iterate over multiple elements if needed to handle all outputs
    for (int idx = thread_idx; idx < batch_size * channels * output_d * output_h * output_w; idx += total_threads) {
        // Reorder index calculation to prioritize width dimension for coalesced access
        const int w_out = idx % output_w;
        const int h_out = (idx / output_w) % output_h;
        const int d_out = (idx / (output_w * output_h)) % output_d;
        const int c = (idx / (output_w * output_h * output_d)) % channels;
        const int b = idx / (output_w * output_h * output_d * channels);

    const int d_start = d_out * stride - padding;
    const int h_start = h_out * stride - padding;
    const int w_start = w_out * stride - padding;

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    int max_index = -1;

    #pragma unroll
    for (int k_d = 0; k_d < kernel_size; k_d++) {
        const int d_in = d_start + k_d * dilation;
        if (d_in < 0 || d_in >= input_d) continue;

        #pragma unroll
        for (int k_h = 0; k_h < kernel_size; k_h++) {
            const int h_in = h_start + k_h * dilation;
            if (h_in < 0 || h_in >= input_h) continue;

            #pragma unroll
            for (int k_w = 0; k_w < kernel_size; k_w++) {
                const int w_in = w_start + k_w * dilation;
                if (w_in < 0 || w_in >= input_w) continue;

                const int input_idx = ((b * channels + c) * input_d + d_in) * input_h * input_w +
                                    h_in * input_w + w_in;
                const scalar_t val = input[input_idx];

                if (val > max_val) {
                    max_val = val;
                    max_index = input_idx;
                }
            }
        }
    }

    output[idx] = max_val;
    if (indices != nullptr) {
        indices[idx] = max_index;
    }
}

// Wrapper function for calling the CUDA kernel
torch::Tensor max_pool3d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation,
    bool return_indices,
    bool ceil_mode) {
    
    auto input_sizes = input.sizes();
    const int batch_size = input_sizes[0];
    const int channels = input_sizes[1];
    const int input_d = input_sizes[2];
    const int input_h = input_sizes[3];
    const int input_w = input_sizes[4];

    const int output_d = ceil_mode ? 
        ceil((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    const int output_h = ceil_mode ?
        ceil((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    const int output_w = ceil_mode ?
        ceil((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);

    auto output = torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options());
    auto indices = return_indices ? 
        torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options().dtype(torch::kLong)) :
        torch::Tensor();

    const int threads = 128;  // Optimized block size
    const int blocks = (batch_size * channels * output_d * output_h * output_w + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool3d_forward_cuda", ([&] {
        max_pool3d_forward_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            return_indices ? indices.data_ptr<int64_t>() : nullptr,
            batch_size, channels,
            input_d, input_h, input_w,
            output_d, output_h, output_w,
            kernel_size, stride, padding, dilation);
    }));

    if (return_indices) {
        return torch::stack({output, indices}, 0);
    }
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool3d_cuda_forward, "Max Pool 3D forward (CUDA)");
}