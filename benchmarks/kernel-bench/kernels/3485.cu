#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Device GELU function specializations
template <typename scalar_t>
__device__ inline scalar_t gelu_function(scalar_t x);

template <>
__device__ inline float gelu_function<float>(float x) {
    return x * 0.5f * (1.0f + erff(x * 0.7071067811865475f));
}

template <>
__device__ inline double gelu_function<double>(double x) {
    return x * 0.5 * (1.0 + erf(x * 0.7071067811865475));
}

// Optimized indexing kernel
// Using 2D grid and block configuration to improve memory coalescing and load balancing

template <typename scalar_t, int VEC_SIZE>
__global__ void gelu_optimized_kernel(const scalar_t* __restrict__ input,
                                       scalar_t* __restrict__ output,
                                       size_t n) {
    const int tid = blockIdx.x * blockDim.x * VEC_SIZE + threadIdx.x * VEC_SIZE;
    const int yid = blockIdx.y * blockDim.y + threadIdx.y;

    int global_tid = tid + yid * gridDim.x * blockDim.x * VEC_SIZE;

    if (global_tid < n) {
        #pragma unroll
        for (int i = 0; i < VEC_SIZE; ++i) {
            scalar_t val = input[global_tid + i];
            output[global_tid + i] = gelu_function(val);
        }
    } else if (global_tid < n) {
        for (int i = 0; i < VEC_SIZE; ++i) {
            int index = global_tid + i;
            if (index < n) {
                scalar_t val = input[index];
                output[index] = gelu_function(val);
            }
        }
    }
}

torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
    auto y = torch::empty_like(x);
    const size_t n = x.numel();
    
    constexpr int VEC_SIZE = 4;
    const int threads_x = 32;
    const int threads_y = 8;
    const dim3 threads(threads_x, threads_y);
    const dim3 blocks((n + threads_x * VEC_SIZE - 1) / (threads_x * VEC_SIZE),
                      (n + threads_y - 1) / threads_y);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "gelu_forward", [&] {
        gelu_optimized_kernel<scalar_t, VEC_SIZE>
            <<<blocks, threads>>>(x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), n);
    });

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA error: ", hipGetErrorString(err));
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "GELU forward (CUDA)");
}
