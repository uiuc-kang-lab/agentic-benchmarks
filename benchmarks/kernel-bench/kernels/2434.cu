#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Define tile sizes
#define BLOCK_SIZE_M 32  // Output tile height (each block computes 32 rows)
#define BLOCK_SIZE_N 16  // Output tile width (each block computes 16 columns)
#define BLOCK_SIZE_K 16  // Reduction tile depth

// Kernel: each thread computes a 2x1 sub-tile (2 rows, 1 column) of C
// A is (K x M): element A[k, m] = A[k * M + m]
// B is (N x K): element B[n, k] = B[n * K + k]
// C is (M x N): element C[m, n] = C[m * N + n]

template <typename scalar_t>
__global__ void matmul_transpose_multi_output_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ C,
    int M,
    int N,
    int K) {

    // Determine the starting indices for this block's tile in C
    int m_start = blockIdx.y * BLOCK_SIZE_M;  // row start in C
    int n_start = blockIdx.x * BLOCK_SIZE_N;  // col start in C

    // Thread indices within the block
    int tx = threadIdx.x; // Expected range: [0, 15]
    int ty = threadIdx.y; // Expected range: [0, 15]

    // Each thread computes two rows: row0 and row1
    int row0 = m_start + tx;             // first row computed by this thread
    int row1 = row0 + (BLOCK_SIZE_M / 2);  // second row computed (offset by 16)
    int col = n_start + ty;              // column index in C

    // Accumulators for the two output elements
    scalar_t acc0 = 0;
    scalar_t acc1 = 0;

    // Declare shared memory tiles
    __shared__ scalar_t A_tile[BLOCK_SIZE_K][BLOCK_SIZE_M]; // Size: 16 x 32
    __shared__ scalar_t B_tile[BLOCK_SIZE_N][BLOCK_SIZE_K];   // Size: 16 x 16

    // Total threads in a block
    int tId = threadIdx.y * blockDim.x + threadIdx.x; // Range: 0 to 255
    int blockSize = blockDim.x * blockDim.y;            // = 256

    int numTiles = (K + BLOCK_SIZE_K - 1) / BLOCK_SIZE_K;
    for (int tile = 0; tile < numTiles; tile++) {
        // Load A tile into shared memory
        // A tile dimensions: BLOCK_SIZE_K x BLOCK_SIZE_M (16 x 32 = 512 elements)
        int totalAElements = BLOCK_SIZE_K * BLOCK_SIZE_M; // 512
        for (int idx = tId; idx < totalAElements; idx += blockSize) {
            int kd = idx / BLOCK_SIZE_M;  // k-index within the tile
            int md = idx % BLOCK_SIZE_M;  // m-index within the tile
            int global_m = m_start + md;  // global m index
            int global_k = tile * BLOCK_SIZE_K + kd;  // global k index
            if (global_m < M && global_k < K)
                A_tile[kd][md] = A[global_k * M + global_m];
            else
                A_tile[kd][md] = 0;
        }

        // Load B tile into shared memory
        // B tile dimensions: BLOCK_SIZE_N x BLOCK_SIZE_K (16 x 16 = 256 elements)
        int totalBElements = BLOCK_SIZE_N * BLOCK_SIZE_K; // 256
        for (int idx = tId; idx < totalBElements; idx += blockSize) {
            int nd = idx / BLOCK_SIZE_K;  // n-index within the tile
            int kd = idx % BLOCK_SIZE_K;  // k-index within the tile
            int global_n = n_start + nd;  // global n index
            int global_k = tile * BLOCK_SIZE_K + kd;  // global k index
            if (global_n < N && global_k < K)
                B_tile[nd][kd] = B[global_n * K + global_k];
            else
                B_tile[nd][kd] = 0;
        }

        __syncthreads();

        // Compute the partial results for this tile
        for (int k = 0; k < BLOCK_SIZE_K; k++) {
            scalar_t a_val0 = A_tile[k][tx];                     // for row0
            scalar_t a_val1 = A_tile[k][tx + (BLOCK_SIZE_M / 2)];  // for row1
            scalar_t b_val = B_tile[ty][k];
            acc0 += a_val0 * b_val;
            acc1 += a_val1 * b_val;
        }
        __syncthreads();
    }

    // Write the results to global memory
    if (row0 < M && col < N) {
        C[row0 * N + col] = acc0;
    }
    if (row1 < M && col < N) {
        C[row1 * N + col] = acc1;
    }
}

// PyTorch binding

torch::Tensor matmul_transpose_cuda(torch::Tensor A, torch::Tensor B) {
    // Dimensions:
    // A: (K x M), B: (N x K), therefore C: (M x N)
    int K = A.size(0);
    int M = A.size(1);
    int N = B.size(0);

    auto C = torch::empty({M, N}, A.options());

    // Define block dimensions: use 16x16 threads per block
    dim3 threads(16, 16);
    // Grid dimensions based on tile sizes
    dim3 blocks((N + BLOCK_SIZE_N - 1) / BLOCK_SIZE_N, (M + BLOCK_SIZE_M - 1) / BLOCK_SIZE_M);

    // Create CUDA stream for asynchronous execution
    hipStream_t stream;
    hipStreamCreate(&stream);

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matmul_transpose_multi_output_kernel", ([&] {
        matmul_transpose_multi_output_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, N, K);
    }));

    // Synchronize the stream
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_transpose_cuda, "Matrix multiplication with transposed inputs using multi-output kernel (CUDA)");
}
