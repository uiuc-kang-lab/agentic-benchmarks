#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void cross_entropy_loss_kernel_optimized(
    const float* __restrict__ logits,
    const int64_t* __restrict__ targets,
    float* __restrict__ losses,
    int batch_size,
    int num_classes
)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int block_size = blockDim.x;
    int grid_size = gridDim.x * block_size;

    // Process multiple elements per thread using grid-stride loop
    for (int i = bid * block_size + tid; i < batch_size; i += grid_size) {
        const float* logits_i = logits + i * num_classes;
        int64_t target = targets[i];

        // Calculate max logit
        float max_logit = logits_i[0];
        for (int j = 1; j < num_classes; ++j) {
            max_logit = fmaxf(max_logit, logits_i[j]);
        }

        // Calculate sum of exp(logits - max_logit)
        float sum_exp = 0.0f;
        for (int j = 0; j < num_classes; ++j) {
            sum_exp += expf(logits_i[j] - max_logit);
        }

        // Calculate loss
        float log_sum_exp = logf(sum_exp);
        losses[i] = -(logits_i[target] - max_logit - log_sum_exp);
    }
}

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.is_cuda(), "predictions must be a CUDA tensor");
    TORCH_CHECK(targets.is_cuda(), "targets must be a CUDA tensor");
    TORCH_CHECK(predictions.dim() == 2, "predictions must be a 2D tensor");
    TORCH_CHECK(targets.dim() == 1, "targets must be a 1D tensor");

    const int batch_size = predictions.size(0);
    const int num_classes = predictions.size(1);
    auto losses = torch::empty({batch_size}, predictions.options());

    // Configure optimal block and grid sizes
    const int threads = 256;
    const int blocks = (batch_size + threads - 1) / threads;

    cross_entropy_loss_kernel_optimized<<<blocks, threads>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<int64_t>(),
        losses.data_ptr<float>(),
        batch_size,
        num_classes
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA error: ", hipGetErrorString(err));
    
    return losses.mean();
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized CrossEntropyLoss forward (CUDA)");
}
