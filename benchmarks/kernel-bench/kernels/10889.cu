#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

struct Float4 { float x, y, z, w; };
struct Bool4 { bool x, y, z, w; };

template <typename scalar_t>
__global__ void masked_cumsum_kernel(
    const scalar_t* __restrict__ x,
    const bool* __restrict__ mask,
    scalar_t* __restrict__ output,
    int64_t chunk_start,
    int64_t n_rows,
    int64_t L) {

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_rows) return;

    const int64_t row = chunk_start + idx;
    scalar_t* output_row = output + row * L;
    const scalar_t* x_row = x + row * L;
    const bool* mask_row = mask + row * L;

    scalar_t sum = 0;
    
    // Vectorized processing - process 4 elements per iteration
    const int64_t vec_steps = L / 4;
    for (int64_t i = 0; i < vec_steps; ++i) {
        const Float4 vals = reinterpret_cast<const Float4*>(x_row)[i];
        const Bool4 masks = reinterpret_cast<const Bool4*>(mask_row)[i];

        if (masks.x) sum += vals.x;
        output_row[i*4] = sum;
        if (masks.y) sum += vals.y;
        output_row[i*4+1] = sum;
        if (masks.z) sum += vals.z;
        output_row[i*4+2] = sum;
        if (masks.w) sum += vals.w;
        output_row[i*4+3] = sum;
    }

    // Process remaining elements
    for (int64_t i = vec_steps*4; i < L; ++i) {
        if (mask_row[i]) {
            sum += x_row[i];
        }
        output_row[i] = sum;
    }
}

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor masked_cumsum(
    const torch::Tensor& x,
    const torch::Tensor& mask,
    int64_t dim) {

    CHECK_INPUT(x);
    CHECK_INPUT(mask);
    TORCH_CHECK(x.sizes() == mask.sizes(), "x and mask must have the same shape");
    TORCH_CHECK(x.is_contiguous(), "Input tensor must be contiguous");

    if (dim < 0) dim += x.dim();
    TORCH_CHECK(dim >= 0 && dim < x.dim(), "Invalid dimension");

    auto x_permuted = x.transpose(dim, -1).contiguous();
    auto mask_permuted = mask.transpose(dim, -1).contiguous();

    const int64_t N = x_permuted.numel() / x_permuted.size(-1);
    const int64_t L = x_permuted.size(-1);

    auto output_flat = torch::empty_like(x_permuted);

    constexpr int num_streams = 4;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    const int64_t chunk_size = (N + num_streams - 1) / num_streams;
    const int threads = 128;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "masked_cumsum", [&] {
        for (int s = 0; s < num_streams; ++s) {
            const int64_t start = s * chunk_size;
            const int64_t end = std::min(start + chunk_size, N);
            const int64_t n_rows = end - start;
            if (n_rows <= 0) continue;

            const int blocks = (n_rows + threads - 1) / threads;
            masked_cumsum_kernel<<<blocks, threads, 0, streams[s]>>>(
                x_permuted.data_ptr<scalar_t>(),
                mask_permuted.data_ptr<bool>(),
                output_flat.data_ptr<scalar_t>(),
                start,
                n_rows,
                L
            );
        }
    });

    // Synchronize streams
    for (int s = 0; s < num_streams; ++s) {
        hipStreamSynchronize(streams[s]);
        hipStreamDestroy(streams[s]);
    }

    auto result = output_flat.transpose(dim, -1);
    return result.contiguous();
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &masked_cumsum, "Vectorized masked cumsum with stream pipelining");
}
