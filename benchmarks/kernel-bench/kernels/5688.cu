#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <limits>

// Each thread processes one output element with even distribution of workload for balanced utilization.
template <typename scalar_t, int KERNEL_SIZE>
__global__ void max_pool2d_balanced_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int stride,
    const int padding,
    const int dilation
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_elements = batch_size * channels * output_height * output_width;
    const int num_threads = gridDim.x * blockDim.x;

    while (index < total_elements) {
        const int ow = index % output_width;
        const int oh = (index / output_width) % output_height;
        const int c = (index / (output_width * output_height)) % channels;
        const int b = index / (output_width * output_height * channels);

        const int input_base = b * channels * input_height * input_width + c * input_height * input_width;
        scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

        #pragma unroll
        for (int kh = 0; kh < KERNEL_SIZE; kh++) {
            const int ih = oh * stride - padding + kh * dilation;
            if (ih >= 0 && ih < input_height) {
                #pragma unroll
                for (int kw = 0; kw < KERNEL_SIZE; kw++) {
                    const int iw = ow * stride - padding + kw * dilation;
                    if (iw >= 0 && iw < input_width) {
                        max_val = fmaxf(max_val, __ldg(&input[input_base + ih * input_width + iw]));
                    }
                }
            }
        }

        output[index] = max_val;
        index += num_threads;  // move index by total number of processing threads for balanced mapping
    }
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const int output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const int output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int total_elements = batch_size * channels * output_height * output_width;
    const int threads = 256;
    const int blocks = (total_elements + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool2d_cuda_forward", ([&] {
        if (kernel_size == 2) {
            max_pool2d_balanced_kernel<scalar_t, 2><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size,
                channels,
                input_height,
                input_width,
                output_height,
                output_width,
                stride,
                padding,
                dilation
            );
        } else if (kernel_size == 3) {
            max_pool2d_balanced_kernel<scalar_t, 3><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size,
                channels,
                input_height,
                input_width,
                output_height,
                output_width,
                stride,
                padding,
                dilation
            );
        } else {
            max_pool2d_balanced_kernel<scalar_t, -1><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size,
                channels,
                input_height,
                input_width,
                output_height,
                output_width,
                stride,
                padding,
                dilation
            );
        }
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Balanced Max Pool 2D forward (CUDA)");
}