#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/util/Optional.h>

namespace py = pybind11;

// -----------------------------------------------------
// Naive 1D convolution CUDA kernel with optional bias
// -----------------------------------------------------
__global__ void conv1d_forward_kernel(
    const float* __restrict__ x,
    const float* __restrict__ w,
    const float* __restrict__ bias_ptr, // can be null if no bias
    float* __restrict__ y,
    const int N,         // batch size
    const int C_in,      // input channels
    const int L_in,      // input length
    const int C_out,     // output channels
    const int K,         // kernel size
    const int stride,
    const int padding,
    const int dilation,
    const int groups,
    const int L_out      // output length
)
{
    // Each thread computes one output element: (n, out_ch, out_pos).
    // Flatten all positions as: idx = n * (C_out * L_out) + out_ch * L_out + out_pos.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = N * C_out * L_out;
    if (idx >= total) return;

    int out_pos = idx % L_out;
    int out_ch  = (idx / L_out) % C_out;
    int n       = idx / (L_out * C_out);

    // Determine group index for this output channel
    int group_size_out = C_out / groups;  
    int group_size_in  = C_in  / groups;  
    int group_idx      = out_ch / group_size_out;

    float val = 0.0f;
    // Convolution accumulation
    for (int local_in_ch = 0; local_in_ch < group_size_in; local_in_ch++) {
        int in_ch = group_idx * group_size_in + local_in_ch;
        for (int k = 0; k < K; k++) {
            int in_pos = out_pos * stride + k * dilation - padding;
            if (in_pos >= 0 && in_pos < L_in) {
                float x_val = x[n * (C_in * L_in) + in_ch * L_in + in_pos];
                float w_val = w[out_ch * (group_size_in * K)
                                + local_in_ch * K
                                + k];
                val += x_val * w_val;
            }
        }
    }

    // Add bias if provided
    if (bias_ptr) {
        val += bias_ptr[out_ch];
    }

    // Store result
    y[n * (C_out * L_out) + out_ch * L_out + out_pos] = val;
}

// -------------------------------------------------------
// Implementation of conv1d forward with optional bias
// -------------------------------------------------------
at::Tensor conv1d_forward_impl(
    const at::Tensor& x,
    const at::Tensor& weight,
    c10::optional<at::Tensor> bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups
)
{
    // Check device, dtype
    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.scalar_type() == at::kFloat, "x must be float32");
    TORCH_CHECK(weight.scalar_type() == at::kFloat, "weight must be float32");

    // x shape: [N, C_in, L_in]
    auto x_sizes = x.sizes();
    int64_t N    = x_sizes[0];
    int64_t C_in = x_sizes[1];
    int64_t L_in = x_sizes[2];

    // weight shape: [C_out, C_in/groups, K]
    auto w_sizes = weight.sizes();
    int64_t C_out = w_sizes[0];
    int64_t K     = w_sizes[2];

    // Calculate conv1d output length
    int64_t L_out = (L_in + 2 * padding - dilation * (K - 1) - 1) / stride + 1;
    TORCH_CHECK(L_out > 0, "Calculated output length is non-positive.");

    // Create output tensor
    auto y = torch::empty({N, C_out, L_out}, x.options().dtype(at::kFloat));

    // Bias pointer (may be null if bias is not provided)
    const float* bias_ptr = nullptr;
    if (bias_opt.has_value() && bias_opt.value().defined()) {
        TORCH_CHECK(bias_opt.value().is_cuda(), "bias must be a CUDA tensor if provided");
        TORCH_CHECK(bias_opt.value().scalar_type() == at::kFloat, "bias must be float32");
        bias_ptr = bias_opt.value().data_ptr<float>();
    }

    // Launch kernel
    int total_threads = N * C_out * L_out;
    int blockSize = 256;
    int gridSize  = (total_threads + blockSize - 1) / blockSize;

    conv1d_forward_kernel<<<gridSize, blockSize>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        y.data_ptr<float>(),
        N, (int)C_in, (int)L_in, (int)C_out, (int)K,
        (int)stride, (int)padding, (int)dilation, (int)groups, 
        (int)L_out
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "conv1d_forward_kernel failed: ", hipGetErrorString(err));

    return y;
}

// -----------------------------------------------------
// Pybind11 binding with optional bias
// -----------------------------------------------------
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        [](at::Tensor x,
           at::Tensor weight,
           py::object bias_obj,
           int64_t stride,
           int64_t padding,
           int64_t dilation,
           int64_t groups) {
            c10::optional<at::Tensor> bias;
            if (!bias_obj.is_none()) {
                bias = bias_obj.cast<at::Tensor>();
            }
            return conv1d_forward_impl(x, weight, bias, stride, padding, dilation, groups);
        },
        "Naive 1D Convolution forward (CUDA) with optional bias"
    );
}