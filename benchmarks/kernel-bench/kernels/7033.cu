#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stdexcept>

template <typename scalar_t>
__global__ void argmin_kernel(const scalar_t* __restrict__ x,
                             int64_t* __restrict__ output,
                             int K,
                             int64_t inner_size,
                             int64_t outer_size) {
    const int inner = blockIdx.x * blockDim.x + threadIdx.x;
    const int outer = blockIdx.y;
    
    if (inner >= inner_size || outer >= outer_size) return;

    const scalar_t* slice_ptr = x + ((int64_t)outer * K * inner_size) + inner;
    scalar_t min_val = __ldg(slice_ptr);
    int min_idx = 0;

    for (int k = 1; k < K; ++k) {
        const scalar_t val = __ldg(slice_ptr + k * inner_size);
        const bool cond = val < min_val;
        min_idx = cond ? k : min_idx;
        min_val = cond ? val : min_val;
    }

    output[(int64_t)outer * inner_size + inner] = min_idx;
}

at::Tensor argmin_cuda_forward(const at::Tensor& x, int64_t dim) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
    const int dims = x.dim();
    const int64_t dim_adj = dim < 0 ? dim + dims : dim;
    TORCH_CHECK(dim_adj >= 0 && dim_adj < dims, "Reduction dim out of range");

    int64_t outer_size = 1;
    for (int i = 0; i < dim_adj; ++i) outer_size *= x.size(i);
    const int K = x.size(dim_adj);
    int64_t inner_size = 1;
    for (int i = dim_adj+1; i < dims; ++i) inner_size *= x.size(i);

    std::vector<int64_t> out_shape;
    for (int i = 0; i < dims; ++i) if (i != dim_adj) out_shape.push_back(x.size(i));
    auto output = at::empty(out_shape, x.options().dtype(at::kLong));

    const int threads = 256;
    const dim3 blocks((inner_size + threads-1)/threads, outer_size);

    AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, x.scalar_type(), "argmin_cuda", [&] {
        argmin_kernel<scalar_t><<<blocks, threads>>>(
            x.data_ptr<scalar_t>(),
            output.data_ptr<int64_t>(),
            K,
            inner_size,
            outer_size
        );
    });

    TORCH_CHECK(hipGetLastError() == hipSuccess, "Kernel launch failed");
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &argmin_cuda_forward, "Argmin forward (CUDA)");
}