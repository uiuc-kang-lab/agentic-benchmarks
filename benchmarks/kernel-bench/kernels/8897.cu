#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <c10/util/Optional.h>

__global__ void conv_transpose2d_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int batch,
    const int in_channels,
    const int in_h,
    const int in_w,
    const int out_channels,
    const int out_h,
    const int out_w,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_h,
    const int pad_w,
    const int dilation_h,
    const int dilation_w,
    const int groups,
    const int in_channels_per_group,
    const int out_channels_per_group) {

    const unsigned int warp_size = 32;
    const unsigned int lane_id = threadIdx.x % warp_size;
    const unsigned int warp_id = threadIdx.x / warp_size;
    
    int index = (blockIdx.x * blockDim.x + threadIdx.x);
    int total = batch * out_channels * out_h * out_w;
    if (index >= total) return;

    // Decode flat index into (n, oc, oh, ow)
    int ow = index % out_w;
    int tmp = index / out_w;
    int oh = tmp % out_h;
    tmp = tmp / out_h;
    int oc = tmp % out_channels;
    int n = tmp / out_channels;

    // Determine which group this output channel belongs to
    int g = oc / out_channels_per_group;
    float out_val = bias[oc];
    
    // Each warp handles a portion of the input channels
    for (int c = g * in_channels_per_group + lane_id; 
         c < (g + 1) * in_channels_per_group; 
         c += warp_size) {
        
        float partial_sum = 0.0f;
        
        // Loop over kernel dimensions
        for (int kh = 0; kh < kernel_h; kh++) {
            int h_in_candidate = oh + pad_h - kh * dilation_h;
            if (h_in_candidate < 0 || (h_in_candidate % stride_h) != 0) continue;
            int ih = h_in_candidate / stride_h;
            if (ih >= in_h) continue;

            for (int kw = 0; kw < kernel_w; kw++) {
                int w_in_candidate = ow + pad_w - kw * dilation_w;
                if (w_in_candidate < 0 || (w_in_candidate % stride_w) != 0) continue;
                int iw = w_in_candidate / stride_w;
                if (iw >= in_w) continue;

                int x_index = n * (in_channels * in_h * in_w) +
                             c * (in_h * in_w) +
                             ih * in_w + iw;

                int weight_index = c * (out_channels_per_group * kernel_h * kernel_w) +
                                  (oc - g * out_channels_per_group) * (kernel_h * kernel_w) +
                                  kh * kernel_w + kw;

                if (c < (g + 1) * in_channels_per_group) {
                    partial_sum += x[x_index] * weight[weight_index];
                }
            }
        }

        // Warp reduction using shuffle
        #pragma unroll
        for (int offset = warp_size/2; offset > 0; offset /= 2) {
            partial_sum += __shfl_down_sync(0xffffffff, partial_sum, offset);
        }

        // First thread in warp has the sum
        if (lane_id == 0) {
            atomicAdd(&out_val, partial_sum);
        }
    }

    // Write final result
    if (lane_id == 0) {
        int out_index = n * (out_channels * out_h * out_w) +
                        oc * (out_h * out_w) +
                        oh * out_w + ow;
        output[out_index] = out_val;
    }
}

at::Tensor forward(
    at::Tensor x,
    at::Tensor weight,
    c10::optional<at::Tensor> bias,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> dilation,
    int groups) {
    
    x = x.contiguous();
    weight = weight.contiguous();
    if (bias.has_value() && bias.value().defined())
        bias = bias.value().contiguous();

    const int batch = x.size(0);
    const int in_channels = x.size(1);
    const int in_h = x.size(2);
    const int in_w = x.size(3);

    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);
    const int out_channels_per_group = weight.size(1);
    const int out_channels = out_channels_per_group * groups;

    const int stride_h = stride[0];
    const int stride_w = stride[1];
    const int pad_h = padding[0];
    const int pad_w = padding[1];
    const int dilation_h = dilation[0];
    const int dilation_w = dilation[1];

    const int out_h = (in_h - 1) * stride_h - 2 * pad_h + dilation_h * (kernel_h - 1) + 1;
    const int out_w = (in_w - 1) * stride_w - 2 * pad_w + dilation_w * (kernel_w - 1) + 1;

    if (!bias.has_value() || !bias.value().defined()) {
        bias = at::zeros({out_channels}, weight.options());
    }

    auto output = at::zeros({batch, out_channels, out_h, out_w}, x.options());

    int in_channels_per_group = in_channels / groups;

    int total_threads = batch * out_channels * out_h * out_w;
    const int threads = 256;
    const int blocks = (total_threads + threads - 1) / threads;

    conv_transpose2d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.value().data_ptr<float>(),
        output.data_ptr<float>(),
        batch,
        in_channels,
        in_h,
        in_w,
        out_channels,
        out_h,
        out_w,
        kernel_h,
        kernel_w,
        stride_h,
        stride_w,
        pad_h,
        pad_w,
        dilation_h,
        dilation_w,
        groups,
        in_channels_per_group,
        out_channels_per_group
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "2D Transposed Convolution (CUDA)",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride"),
          py::arg("padding"),
          py::arg("dilation"),
          py::arg("groups"));
}