#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel for ReLU activation with optimized synchronization
template <typename scalar_t>
__global__ void relu_kernel_optimized(
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ input,
    const int64_t size) {
    
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size; idx += blockDim.x * gridDim.x) {
        const scalar_t val = input[idx];
        output[idx] = val > 0 ? val : 0;
    }
}

// PyTorch wrapper function
torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    
    const int threads = 256;
    const int blocks = (input.numel() + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "relu_kernel_optimized", ([&] {
        relu_kernel_optimized<scalar_t><<<blocks, threads>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            input.numel()
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ReLU forward optimized (CUDA)");
}