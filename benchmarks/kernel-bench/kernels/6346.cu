#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel using warp-level primitives with uniform control flow to minimize warp divergence.
// Each warp computes one output element by summing over a segment of the reduction dimension.
// The reduction within each warp is performed using __shfl_down_sync.

template <typename scalar_t>
__global__ void uniform_control_flow_warp_reduce_sum_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t reduce_size,
    int64_t inner_size,
    int64_t total_outputs) {

    const int warpSize = 32;
    int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = global_thread_id / warpSize;
    int lane = global_thread_id % warpSize;
    int total_warps = (gridDim.x * blockDim.x) / warpSize;

    // Each warp processes output elements in a grid-stride loop
    for (int out_idx = warp_id; out_idx < total_outputs; out_idx += total_warps) {
        int outer_idx = out_idx / inner_size;
        int inner_idx = out_idx % inner_size;

        int64_t base = outer_idx * reduce_size * inner_size + inner_idx;
        scalar_t sum = 0;

        // Uniform control flow: each thread processes its portion of the reduction dimension
        for (int i = lane; i < reduce_size; i += warpSize) {
            sum += input[base + i * inner_size];
        }

        // Warp-level reduction with uniform control flow
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
        }

        // First lane writes the result
        if (lane == 0) {
            output[out_idx] = sum;
        }
    }
}

// CUDA wrapper function
torch::Tensor sum_reduce_cuda(torch::Tensor input, int64_t dim) {
    // Adjust for negative dimensions
    if (dim < 0) dim += input.dim();

    auto sizes = input.sizes().vec();
    int64_t reduce_size = sizes[dim];

    // Compute outer and inner dimensions
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }
    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }

    // Set the reduced dimension to 1
    sizes[dim] = 1;
    auto output = torch::empty(sizes, input.options());

    // Total number of output elements
    int64_t total_outputs = outer_size * inner_size;

    // Each output element is computed by one warp (32 threads)
    const int warpSize = 32;
    int total_threads = total_outputs * warpSize;
    int threads = 256;  // Must be a multiple of 32
    int blocks = (total_threads + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sum_reduce_cuda", ([&] {
        uniform_control_flow_warp_reduce_sum_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            reduce_size,
            inner_size,
            total_outputs
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &sum_reduce_cuda, "Sum reduction forward (CUDA) with uniform control flow");
}
