#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>

// Device function to compute the max pooling result for one output element
// Using __restrict__ to optimize memory access
// Use of __forceinline__ for potential performance improvement
// Precompute input_plane and offset to reduce redundant calculations
template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_maxpool(
    const scalar_t* __restrict__ input,
    int b, int c, int oh, int ow,
    int input_height, int input_width,
    int kernel_size, int stride, int padding, int dilation,
    int channels) {

  scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
  int input_plane = input_height * input_width;
  int offset = b * channels * input_plane + c * input_plane;

  for (int kh = 0; kh < kernel_size; ++kh) {
    for (int kw = 0; kw < kernel_size; ++kw) {
      int ih = oh * stride - padding + kh * dilation;
      int iw = ow * stride - padding + kw * dilation;
      if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
        int in_idx = offset + ih * input_width + iw;
        scalar_t val = input[in_idx];
        if (val > max_val) {
          max_val = val;
        }
      }
    }
  }
  return max_val;
}

// Kernel that uses a grid-stride loop and the modular device function
// Incorporates error checking directly after kernel call
// Uses grid-stride loop for better GPU utilization
// __restrict__ pointers for improved memory access patterns
// Error checking moved directly after kernel launch
template <typename scalar_t>
__global__ void optimized_maxpool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation) {

  const int total = batch_size * channels * output_height * output_width;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int gridSize = blockDim.x * gridDim.x;

  for (; idx < total; idx += gridSize) {
    int ow = idx % output_width;
    int oh = (idx / output_width) % output_height;
    int c  = (idx / (output_width * output_height)) % channels;
    int b  = idx / (output_width * output_height * channels);

    scalar_t max_val = compute_maxpool(input, b, c, oh, ow,
                                         input_height, input_width,
                                         kernel_size, stride, padding, dilation,
                                         channels);
    output[idx] = max_val;
  }
}

// Host function to prepare parameters and launch the kernel
// Error checking moved directly after kernel launch
// Use of AT_DISPATCH_FLOATING_TYPES for flexibility
// Ensures the input tensor is CUDA and prepares output tensor
// Calculates output dimensions and launches optimized kernel
// Improved error handling

torch::Tensor optimized_modular_maxpool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation) {
  TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");

  const int batch_size = input.size(0);
  const int channels = input.size(1);
  const int input_height = input.size(2);
  const int input_width = input.size(3);

  int output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
  int output_width  = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

  auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

  int total_elements = batch_size * channels * output_height * output_width;
  const int threads = 256;
  int blocks = (total_elements + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "optimized_modular_maxpool2d_cuda_forward", ([&] {
    optimized_maxpool2d_kernel<scalar_t><<<blocks, threads>>>(
        input.data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>(),
        batch_size,
        channels,
        input_height,
        input_width,
        output_height,
        output_width,
        kernel_size,
        stride,
        padding,
        dilation);
  }));

  hipError_t err = hipGetLastError();
  TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &optimized_modular_maxpool2d_cuda_forward, "Optimized Modular Max Pool 2D forward (CUDA)");
}
