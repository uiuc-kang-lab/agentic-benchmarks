#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__constant__ float const_weight[2048]; // Presuming we use up to 2048 elements, ensure this matches the maximum expected size
__constant__ float const_bias[128]; // Presuming up to 128 bias terms, adjust as necessary

__device__ inline float process_kernel_element(
    const float* x,
    int base_x,
    int base_w,
    int k,
    int input_pos,
    int in_size
) {
    if (input_pos >= 0 && input_pos < in_size) {
        return x[base_x + input_pos] * const_weight[base_w + k];
    }
    return 0.0f;
}

__device__ float process_input_channel(
    const float* x,
    int b,
    int oc,
    int o,
    int ic,
    int in_channels,
    int in_size,
    int kernel_size,
    int stride,
    int dilation
) {
    float channel_sum = 0.0f;
    const int base_x = b * (in_channels * in_size) + ic * in_size;
    const int base_w = oc * (in_channels * kernel_size) + ic * kernel_size;

    #pragma unroll 4
    for (int k = 0; k < kernel_size; ++k) {
        const int input_pos = o * stride + k * dilation;
        channel_sum += process_kernel_element(x, base_x, base_w, k, input_pos, in_size);
    }
    return channel_sum;
}

__global__ void conv1d_kernel(
    const float* __restrict__ x,
    float* __restrict__ output,
    int B,
    int in_channels,
    int in_size,
    int out_channels,
    int kernel_size,
    int out_size,
    int stride,
    int dilation
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_elements = B * out_channels * out_size;
    if (idx >= total_elements) return;

    const int o = idx % out_size;
    const int oc = (idx / out_size) % out_channels;
    const int b = idx / (out_size * out_channels);

    float sum = 0.0f;

    for (int ic = 0; ic < in_channels; ++ic) {
        sum += process_input_channel(x, b, oc, o, ic,
                                   in_channels, in_size, kernel_size,
                                   stride, dilation);
    }

    if (bias.has_value()) sum += const_bias[oc];
    output[b * (out_channels * out_size) + oc * out_size + o] = sum;
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int dilation
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(weight.dim() == 3, "weight must be 3D");
    TORCH_CHECK(weight.size(1) == x.size(1), "Input channels mismatch");

    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        TORCH_CHECK(bias->dim() == 1, "bias must be 1D");
        TORCH_CHECK(bias->size(0) == weight.size(0), "Bias size mismatch");
    }

    const int B = x.size(0);
    const int in_channels = x.size(1);
    const int in_size = x.size(2);
    const int out_channels = weight.size(0);
    const int kernel_size = weight.size(2);

    const int out_size = (in_size - dilation * (kernel_size - 1) - 1) / stride + 1;
    TORCH_CHECK(out_size > 0, "Invalid output size");

    auto output = torch::empty({B, out_channels, out_size}, x.options());
    if (output.numel() == 0) return output;

    hipMemcpyToSymbol(HIP_SYMBOL(const_weight), weight.data_ptr<float>(), weight.numel() * sizeof(float));
    if (bias.has_value()) {
        hipMemcpyToSymbol(HIP_SYMBOL(const_bias), bias->data_ptr<float>(), bias->numel() * sizeof(float));
    }

    const int total_elements = B * out_channels * out_size;
    constexpr int threads = 256;
    const int blocks = (total_elements + threads - 1) / threads;

    conv1d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        output.data_ptr<float>(),
        B, in_channels, in_size,
        out_channels, kernel_size, out_size,
        stride, dilation
    );

    TORCH_CHECK(hipGetLastError() == hipSuccess, "Kernel launch failed");
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized 1D convolution with constant memory usage");
}