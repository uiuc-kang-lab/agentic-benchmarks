#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <float.h>

// This kernel computes the argmax over a specified dimension.
// It uses __ldg() to optimize read-only global memory loads and assumes that the input is 128-bit aligned.
// Each block handles one slice (an outer-inner pair) and performs a cooperative reduction in shared memory using warp-level operations.

__global__ void argmax_kernel_ldg(
    const float* __restrict__ x,
    int64_t* __restrict__ indices,
    const int outerSize,
    const int dimSize,
    const int innerSize) {

    // Each block processes one slice (one outer-inner pair)
    int slice = blockIdx.x;
    if (slice >= outerSize * innerSize) return;

    int outer_idx = slice / innerSize;
    int inner_idx = slice % innerSize;
    int base_offset = outer_idx * (dimSize * innerSize) + inner_idx;

    // Each thread computes a local maximum for a subset of the reduction dimension
    float local_max = -FLT_MAX;
    int local_argmax = 0;

    // Use __ldg() to load data from global memory (read-only cache), assuming proper alignment
    for (int d = threadIdx.x; d < dimSize; d += blockDim.x) {
        float curr_val = __ldg(&x[base_offset + d * innerSize]);
        if (__ldg(&x[base_offset + d * innerSize]) > local_max) {
            local_max = curr_val;
            local_argmax = d;
        }
    }

    // Shared memory for reduction across threads in a block
    extern __shared__ char shared_mem[];
    float* s_max = reinterpret_cast<float*>(shared_mem);
    int* s_idx = reinterpret_cast<int*>(s_max + blockDim.x);

    s_max[threadIdx.x] = local_max;
    s_idx[threadIdx.x] = local_argmax;
    __syncthreads();

    // Reduction in shared memory: standard tree reduction
    for (unsigned int stride = blockDim.x / 2; stride > 64; stride >>= 1) {
        if (threadIdx.x < stride) {
            if (s_max[threadIdx.x + stride] > s_max[threadIdx.x]) {
                s_max[threadIdx.x] = s_max[threadIdx.x + stride];
                s_idx[threadIdx.x] = s_idx[threadIdx.x + stride];
            }
        }
        __syncthreads();
    }

    // Warp-level reduction without __syncthreads()
    if (threadIdx.x < 64) {
        if (s_max[threadIdx.x + 64] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 64];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 64];
        }
        if (s_max[threadIdx.x + 32] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 32];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 32];
        }
        if (s_max[threadIdx.x + 16] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 16];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 16];
        }
        if (s_max[threadIdx.x + 8] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 8];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 8];
        }
        if (s_max[threadIdx.x + 4] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 4];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 4];
        }
        if (s_max[threadIdx.x + 2] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 2];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 2];
        }
        if (s_max[threadIdx.x + 1] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 1];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 1];
        }
    }

    // Write the result
    if (threadIdx.x == 0) {
        indices[slice] = s_idx[0];
    }
}

// Host function to launch the CUDA kernel

torch::Tensor argmax_forward_cuda(const torch::Tensor& x, const int64_t dim) {
    TORCH_CHECK(x.scalar_type() == at::kFloat, "Only float32 is supported.");
    auto x_contig = x.contiguous();

    auto sizes = x_contig.sizes();
    int ndim = x_contig.dim();
    TORCH_CHECK(dim >= 0 && dim < ndim, "Invalid dim for argmax.");

    // Compute outerSize (product of dims before 'dim'), dimSize, and innerSize (product of dims after 'dim')
    int outerSize = 1;
    for (int d = 0; d < dim; d++) {
        outerSize *= sizes[d];
    }
    int dimSize = sizes[dim];
    int innerSize = 1;
    for (int d = dim + 1; d < ndim; d++) {
        innerSize *= sizes[d];
    }

    // Prepare output shape: remove the reduced dimension
    std::vector<int64_t> out_sizes;
    for (int d = 0; d < ndim; d++) {
        if (d == dim) continue;
        out_sizes.push_back(sizes[d]);
    }
    auto options = torch::TensorOptions().device(x.device()).dtype(torch::kLong);
    auto indices = torch::empty(out_sizes, options);

    // Each slice corresponds to one (outer, inner) pair
    int slices = outerSize * innerSize;

    // Use 128 threads per block for good occupancy; shared memory holds 128 floats and 128 ints
    const int threads = 128;
    int blocks = slices;
    int sharedMemSize = threads * (sizeof(float) + sizeof(int));

    argmax_kernel_ldg<<<blocks, threads, sharedMemSize>>>(
        x_contig.data_ptr<float>(),
        indices.data_ptr<int64_t>(),
        outerSize,
        dimSize,
        innerSize
    );

    return indices;
}

// Pybind11 binding
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &argmax_forward_cuda, "ArgMax CUDA forward using __ldg() for optimized global memory loads");
}
