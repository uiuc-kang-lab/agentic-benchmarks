#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void matvec_mul_kernel_aligned(
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> A,
    const torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> B,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> C,
    int64_t M,
    int64_t K) 
{
    const int warp_size = 32;
    const int warp_id = threadIdx.x / warp_size;
    const int lane = threadIdx.x % warp_size;
    const int warps_per_block = blockDim.x / warp_size;
    
    // Grid-stride loop over rows
    for (int64_t row = blockIdx.x * warps_per_block + warp_id; row < M; row += gridDim.x * warps_per_block) {
        scalar_t sum = 0;
        
        // Ensure 128-bit aligned access when possible
        const scalar_t* row_ptr = &A[row][0];
        
        // Process elements in chunks of 4 when possible for better memory coalescing
        int64_t col;
        #pragma unroll 4
        for (col = lane; col + 3 < K; col += warp_size) {
            // Use __ldg for read-only memory access
            scalar_t a0 = __ldg(row_ptr + col);
            scalar_t b0 = __ldg(&B[col]);
            sum += a0 * b0;
        }
        
        // Handle remaining elements
        if (col < K) {
            sum += __ldg(row_ptr + col) * __ldg(&B[col]);
        }
        
        // Warp-level reduction using shuffle
        #pragma unroll
        for (int offset = warp_size/2; offset > 0; offset >>= 1) {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }
        
        if (lane == 0) {
            C[row][0] = sum;
        }
    }
}

torch::Tensor matvec_mul_cuda(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");

    // Ensure inputs are contiguous and properly aligned
    A = A.contiguous();
    B = B.contiguous();

    int64_t M = A.size(0);
    int64_t K = A.size(1);

    TORCH_CHECK(B.numel() == K, "B must have the same number of elements as columns in A");
    auto B_flat = B.view({-1});
    
    // Ensure output tensor is properly aligned
    auto options = A.options(); // No alignment method available
    auto C = torch::zeros({M, 1}, options);

    const int threads_per_block = 256;
    const int warps_per_block = threads_per_block / 32;
    
    int num_sms;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    int blocks_per_sm = 2;
    int num_blocks = min(
        (static_cast<int>(M) + warps_per_block - 1) / warps_per_block,
        num_sms * blocks_per_sm
    );

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matvec_mul_cuda", ([&] {
        matvec_mul_kernel_aligned<scalar_t><<<num_blocks, threads_per_block>>>(
            A.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
            B_flat.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
            C.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
            M,
            K);
    }));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matvec_mul_cuda, "Aligned Matrix-Vector Multiplication (CUDA)");
}