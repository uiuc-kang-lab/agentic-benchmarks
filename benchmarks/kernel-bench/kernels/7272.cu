#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define TILE_SIZE 32
#define KERNEL_SIZE 3
#define MAX_WEIGHT_SIZE 16384  // Maximum number of floats to store in constant memory (~64KB)

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Declare constant memory for the kernel weights
__constant__ float d_weight_const[MAX_WEIGHT_SIZE];

// CUDA kernel that reads convolution weights from constant memory
__global__ void conv2d_constmem_kernel(
    const float* __restrict__ input,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int out_channels,
    const int out_height,
    const int out_width,
    const int stride,
    const int padding,
    const int dilation) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x * TILE_SIZE;
    int by = blockIdx.y * TILE_SIZE;
    
    // Decode batch and output channel from the third grid dimension
    int b_oc = blockIdx.z;
    int b = b_oc / out_channels;
    int oc = b_oc % out_channels;

    // Compute output indices
    int out_h = by + ty;
    int out_w = bx + tx;

    float sum = (bias != nullptr) ? bias[oc] : 0.0f;

    if (out_h < out_height && out_w < out_width) {
        // Compute base input indices, taking stride and padding into account
        // Incorporate dilation for proper spacing in the kernel window
        int base_in_h = out_h * stride - padding;
        int base_in_w = out_w * stride - padding;
        
        // Loop over all input channels and over the kernel window
        for (int ic = 0; ic < in_channels; ic++) {
            for (int kh = 0; kh < KERNEL_SIZE; kh++) {
                int in_h = base_in_h + kh * dilation;
                if (in_h < 0 || in_h >= in_height) continue;
                for (int kw = 0; kw < KERNEL_SIZE; kw++) {
                    int in_w = base_in_w + kw * dilation;
                    if (in_w < 0 || in_w >= in_width) continue;
                    int input_idx = ((b * in_channels + ic) * in_height + in_h) * in_width + in_w;
                    int weight_idx = ((oc * in_channels + ic) * KERNEL_SIZE + kh) * KERNEL_SIZE + kw;
                    sum += input[input_idx] * d_weight_const[weight_idx];
                }
            }
        }
        int output_idx = ((b * out_channels + oc) * out_height + out_h) * out_width + out_w;
        output[output_idx] = sum;
    }
}

// Host function to copy weights to constant memory and launch the kernel
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    // Retrieve tensor dimensions
    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int in_height = x.size(2);
    int in_width = x.size(3);
    int out_channels = weight.size(0);
    
    int kernel_size = weight.size(2);
    TORCH_CHECK(kernel_size == KERNEL_SIZE, "Kernel size mismatch.");

    int out_height = (in_height + 2 * padding - dilation * (KERNEL_SIZE - 1) - 1) / stride + 1;
    int out_width = (in_width + 2 * padding - dilation * (KERNEL_SIZE - 1) - 1) / stride + 1;

    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, x.options());
    
    // Ensure the weight tensor fits in constant memory
    int weight_elements = weight.numel();
    TORCH_CHECK(weight_elements <= MAX_WEIGHT_SIZE, "Weight tensor is too large for constant memory.");
    
    // Copy weights from the input tensor to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_weight_const), weight.data_ptr<float>(), weight_elements * sizeof(float), 0, hipMemcpyDeviceToDevice);

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks(
        (out_width + TILE_SIZE - 1) / TILE_SIZE,
        (out_height + TILE_SIZE - 1) / TILE_SIZE,
        batch_size * out_channels
    );
    
    conv2d_constmem_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        bias.has_value() ? bias.value().data_ptr<float>() : nullptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        in_height,
        in_width,
        out_channels,
        out_height,
        out_width,
        stride,
        padding,
        dilation
    );

    hipDeviceSynchronize();
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA forward function for 2D convolution using constant memory for weights");
}
