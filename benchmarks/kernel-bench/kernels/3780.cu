#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_softplus(const scalar_t x) {
    if (x > static_cast<scalar_t>(20.0)) {
        return x;
    } else if (x < static_cast<scalar_t>(-20.0)) {
        return exp(x);
    }
    return log1p(exp(x));
}

template <typename scalar_t>
__global__ void softplus_kernel_2d(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int rows,
    const int cols) {
    
    // 2D thread indexing
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < rows && col < cols) {
        const int idx = row * cols + col;
        const scalar_t x = input[idx];
        output[idx] = compute_softplus(x);
    }
}

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    
    // Get input dimensions
    const int size = input.numel();
    const int rows = input.size(0);
    const int cols = size / rows;
    
    // Define 2D grid configuration
    dim3 threads(16, 16);
    dim3 blocks(
        (cols + threads.x - 1) / threads.x,
        (rows + threads.y - 1) / threads.y
    );

    AT_DISPATCH_FLOATING_TYPES(input.type(), "softplus_forward_cuda", ([&] {
        softplus_kernel_2d<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            rows,
            cols);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}