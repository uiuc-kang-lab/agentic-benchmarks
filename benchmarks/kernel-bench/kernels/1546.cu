#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32
#define TILE_SIZE 32  // Aligned with warp size

__global__ void matmul_kernel(const float* __restrict__ A,
                             const float* __restrict__ B,
                             float* __restrict__ C,
                             const int N,
                             const int N_aligned) {
    __shared__ float s_A[TILE_SIZE][TILE_SIZE];
    __shared__ float s_B[TILE_SIZE][TILE_SIZE];

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int row = blockIdx.y * TILE_SIZE + ty;
    const int col = blockIdx.x * TILE_SIZE + tx;
    
    // Pre-compute array indices
    const int row_N = row * N;
    float value = 0.0f;

    // Compute aligned dimensions for uniform execution
    const int num_tiles = (N_aligned + TILE_SIZE - 1) / TILE_SIZE;
    
    for (int t = 0; t < num_tiles; ++t) {
        // Compute tile offset once
        const int tile_offset = t * TILE_SIZE;
        
        // Load data into shared memory
        // Use aligned indices for coalesced access
        const int a_idx = row_N + tile_offset + tx;
        const int b_idx = (tile_offset + ty) * N + col;
        
        // Predicated loads without branching
        s_A[ty][tx] = (row < N && (tile_offset + tx) < N) ? A[a_idx] : 0.0f;
        s_B[ty][tx] = ((tile_offset + ty) < N && col < N) ? B[b_idx] : 0.0f;
        
        __syncthreads();
        
        // Compute tile product without branches
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k += WARP_SIZE) {
            // Process entire warp's worth of elements at once
            #pragma unroll
            for (int w = 0; w < WARP_SIZE; ++w) {
                value = fmaf(s_A[ty][k + w], s_B[k + w][tx], value);
            }
        }
        
        __syncthreads();
    }

    // Write result using predication instead of branching
    if (row < N && col < N) {
        C[row * N + col] = value;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "Inputs must be 2D tensors");
    TORCH_CHECK(A.size(0) == A.size(1) && B.size(0) == B.size(1), "Inputs must be square matrices");
    TORCH_CHECK(A.size(0) == B.size(0), "Input dimensions must match");

    const int N = A.size(0);
    // Align dimensions to warp size for uniform execution
    const int N_aligned = ((N + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;

    auto C = torch::zeros({N, N}, A.options());

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((N_aligned + TILE_SIZE - 1) / TILE_SIZE,
                (N_aligned + TILE_SIZE - 1) / TILE_SIZE);

    matmul_kernel<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N,
        N_aligned
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Warp-aligned Matrix Multiplication (CUDA)");
}