#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

#define TILE_SIZE 16
#define BLOCK_SIZE 256

__global__ void depthwise_conv2d_unroll_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch,
    int channels,
    int in_h,
    int in_w,
    int out_h,
    int out_w,
    int kernel_h,
    int stride,
    int padding,
    int dilation) {
    
    // Calculate tile indices
    int tile_row = blockIdx.y * TILE_SIZE;
    int tile_col = blockIdx.x * TILE_SIZE;
    int channel = blockIdx.z % channels;
    int batch_idx = blockIdx.z / channels;
    
    // Thread indices within the block
    int tx = threadIdx.x % TILE_SIZE;
    int ty = threadIdx.x / TILE_SIZE;
    
    // Global output positions
    int oh = tile_row + ty;
    int ow = tile_col + tx;
    
    if (batch_idx < batch && oh < out_h && ow < out_w) {
        float sum = 0.0f;
        
        // Unroll the loop for kernel height
        #pragma unroll
        for (int kh = 0; kh < kernel_h; ++kh) {
            int ih = oh * stride - padding + kh * dilation;
            int iw = ow * stride - padding;
            
            if (ih >= 0 && ih < in_h && iw >= 0 && iw < in_w) {
                int input_idx = ((batch_idx * channels + channel) * in_h + ih) * in_w + iw;
                int weight_idx = channel * kernel_h + kh;
                sum += input[input_idx] * weight[weight_idx];
            }
        }
        
        // Add bias
        sum += bias[channel];
        
        // Write output
        int output_idx = ((batch_idx * channels + channel) * out_h + oh) * out_w + ow;
        if (oh < out_h && ow < out_w) {
            output[output_idx] = sum;
        }
    }
}

at::Tensor forward(
    at::Tensor x,
    at::Tensor weight,
    c10::optional<at::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {
    
    x = x.contiguous();
    weight = weight.contiguous();
    
    int batch = x.size(0);
    int channels = x.size(1);
    int in_h = x.size(2);
    int in_w = x.size(3);
    int kernel_h = weight.size(2);
    
    if (groups != channels) {
        throw std::invalid_argument("Depthwise convolution requires groups == number of input channels.");
    }
    
    at::Tensor bias_val;
    if (bias.has_value() && bias.value().defined()) {
        bias_val = bias.value().contiguous();
    } else {
        bias_val = at::zeros({channels}, x.options());
    }
    
    int out_h = (in_h + 2 * padding - dilation * (kernel_h - 1) - 1) / stride + 1;
    int out_w = (in_w + 2 * padding - 1) / stride + 1;
    
    auto output = at::empty({batch, channels, out_h, out_w}, x.options());
    
    // Calculate grid dimensions for tiled approach
    dim3 grid(
        (out_w + TILE_SIZE - 1) / TILE_SIZE,
        (out_h + TILE_SIZE - 1) / TILE_SIZE,
        batch * channels
    );
    dim3 block(BLOCK_SIZE);
    
    const float* x_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias_val.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();
    
    depthwise_conv2d_unroll_kernel<<<grid, block>>>(
        x_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch,
        channels,
        in_h,
        in_w,
        out_h,
        out_w,
        kernel_h,
        stride,
        padding,
        dilation
    );
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Depthwise 2D Convolution forward (CUDA)",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = c10::nullopt,
          py::arg("stride"),
          py::arg("padding"),
          py::arg("dilation"),
          py::arg("groups"));
}