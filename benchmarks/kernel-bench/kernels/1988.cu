#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define NUM_STREAMS 4

__global__ void combined_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int N,
    int start_row,
    int end_row
) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y + start_row;

    if (row >= end_row || col >= N) return;

    if (row < col) {
        C[row * N + col] = 0.0f;
    } else {
        float sum = 0.0f;
        for (int k = col; k <= row; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

at::Tensor forward_combined(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    const int N = A.size(0);
    auto C = torch::empty_like(A);

    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamCreate(&streams[i]);
    }

    // Optimized block configuration from Kernel 1
    const int bx = 64, by = 16;
    dim3 threads(bx, by);
    const int chunk_size = (N + NUM_STREAMS - 1) / NUM_STREAMS;

    for (int i = 0; i < NUM_STREAMS; ++i) {
        int start = i * chunk_size;
        int end = std::min(start + chunk_size, N);
        if (start >= end) continue;

        dim3 blocks(
            (N + bx - 1) / bx,
            (end - start + by - 1) / by
        );

        combined_kernel<<<blocks, threads, 0, streams[i]>>>(
            A.data_ptr<float>(),
            B.data_ptr<float>(),
            C.data_ptr<float>(),
            N,
            start,
            end
        );
    }

    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_combined, "Combined stream+optimized block triangular matmul (CUDA)");
}