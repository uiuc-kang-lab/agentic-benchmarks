#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// CUDA kernel with manually unrolled grid-stride loop
__global__ void softsign_kernel_unrolled(const float* __restrict__ x, float* __restrict__ out, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Process 4 elements per iteration using manual unrolling
    for (int i = idx; i < num_elements; i += stride * 4) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            int index = i + j * stride;
            if (index < num_elements) {
                float val = x[index];
                out[index] = val / (1.0f + fabsf(val));
            }
        }
    }
}

// Host function
torch::Tensor forward(torch::Tensor x) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int num_elements = x.numel();
    int threads = 1024;
    int blocks = (num_elements + threads - 1) / threads;

    softsign_kernel_unrolled<<<blocks, threads>>>(x.data_ptr<float>(), out.data_ptr<float>(), num_elements);
    
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Softsign activation with unrolled loops (CUDA)");
}
