#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>

namespace py = pybind11;

#define THREADS_PER_BLOCK 256

template <typename scalar_t>
__global__ void depthwise_conv2d_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    int batch,
    int channels,
    int in_h, int in_w,
    int out_h, int out_w,
    int k,
    int stride,
    int padding,
    int dilation) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch * channels * out_h * out_w;
  if (index >= total)
      return;

  int ow = index % out_w;
  int tmp = index / out_w;
  int oh = tmp % out_h;
  tmp = tmp / out_h;
  int c = tmp % channels;
  int n = tmp / channels;

  scalar_t sum = 0;
  #pragma unroll
  for (int i = 0; i < k; ++i) {
    #pragma unroll
    for (int j = 0; j < k; ++j) {
      int ih = oh * stride - padding + i * dilation;
      int iw = ow * stride - padding + j * dilation;
      if (ih >= 0 && ih < in_h && iw >= 0 && iw < in_w) {
        int input_idx = n * channels * in_h * in_w + c * in_h * in_w + ih * in_w + iw;
        int weight_idx = c * k * k + i * k + j;
        sum += input[input_idx] * weight[weight_idx];
      }
    }
  }
  if (bias != nullptr)
    sum += bias[c];
  output[index] = sum;
}

template <typename scalar_t>
__global__ void pointwise_conv2d_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    int batch,
    int in_channels,
    int out_channels,
    int h,
    int w) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch * out_channels * h * w;
  if (index >= total)
      return;

  int ow = index % w;
  int tmp = index / w;
  int oh = tmp % h;
  tmp = tmp / h;
  int oc = tmp % out_channels;
  int n = tmp / out_channels;

  scalar_t sum = 0;
  #pragma unroll
  for (int ic = 0; ic < in_channels; ++ic) {
    int input_idx = n * in_channels * h * w + ic * h * w + oh * w + ow;
    int weight_idx = oc * in_channels + ic;
    sum += input[input_idx] * weight[weight_idx];
  }
  if (bias != nullptr)
    sum += bias[oc];
  output[index] = sum;
}

// Remainder of the code (forward_cuda, toTensor, forward_wrapper, PYBIND11_MODULE)
// ... Identical to reference implementation ...