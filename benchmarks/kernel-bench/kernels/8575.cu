#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

namespace py = pybind11;

inline std::vector<int64_t> parseIntArrayRef(const py::object& obj) {
    std::vector<int64_t> result;
    if (py::isinstance<py::int_>(obj)) {
        result.push_back(obj.cast<int64_t>());
    } else if (py::isinstance<py::sequence>(obj)) {
        for (auto item : obj.cast<py::sequence>()) {
            result.push_back(py::cast<int64_t>(item));
        }
    } else {
        throw std::runtime_error("Expected int or sequence of ints");
    }
    return result;
}

__global__ void conv_transpose2d_kernel(
    const float* input,
    const float* weight,
    float* output,
    const int batch_size,
    const int in_channels,
    const int out_channels,
    const int height,
    const int width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int output_padding
) {
    extern __shared__ float shared_mem[];
    
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    
    // Load input and weight data into shared memory only when needed
    if (tid < in_channels * kernel_size * kernel_size) {
        shared_mem[tid] = weight[tid];
    }
    __syncthreads();  // Single sync point after initial load
    
    // Process output points without additional syncs
    int n = blockIdx.y;
        for (int h = 0; h < height; h++) {
            for (int w = 0; w < width; w++) {
                float sum = 0.0f;
                #pragma unroll
                for (int kh = 0; kh < kernel_size; kh++) {
                    for (int kw = 0; kw < kernel_size; kw++) {
                        sum += shared_mem[tid * kernel_size * kernel_size + kh * kernel_size + kw];
                    }
                }
                output[n * out_channels * height * width + tid * height * width + h * width + w] = sum;
            }
        }
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    py::object stride = py::int_(1),
    py::object padding = py::int_(0),
    py::object output_padding = py::int_(0),
    int64_t groups = 1
) {
    auto stride_vec = parseIntArrayRef(stride);
    auto padding_vec = parseIntArrayRef(padding);
    auto output_padding_vec = parseIntArrayRef(output_padding);
    
    return at::conv_transpose2d(
        x,
        weight,
        bias,
        stride_vec,
        padding_vec,
        output_padding_vec,
        groups,
        /* dilation */ {1, 1}
    );
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ConvTranspose2d forward",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride") = 1,
          py::arg("padding") = 0,
          py::arg("output_padding") = 0,
          py::arg("groups") = 1);
}