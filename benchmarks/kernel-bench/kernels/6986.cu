#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stdexcept>

// Store frequently used data in constant memory to optimize memory access times.
__constant__ int const_mem_k[1];

// Optimized kernel utilizing constant memory and coalesced memory access.
template <typename scalar_t>
__global__ void optimized_argmin_kernel(const scalar_t* __restrict__ x,
                                        int64_t* __restrict__ output,
                                        int64_t inner_size) {
  // Fetch K from constant memory
  int K = const_mem_k[0];

  // Compute a global thread index (as a 64-bit integer)
  int64_t idx = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  int64_t total_slices = gridDim.y * inner_size;
  if (idx >= total_slices) return;

  // Decompose idx into outer and inner indices
  int64_t outer = idx / inner_size;
  int64_t inner = idx % inner_size;

  // Data layout is interpreted as: [outer, K, inner]
  // Start of the slice for this (outer, inner) location
  const scalar_t* slice_start = x + outer * (static_cast<int64_t>(K) * inner_size) + inner;

  scalar_t min_val = slice_start[0];
  int min_index = 0;
  bool found = true;
  // Iterate over the K dimension with strides of inner_size
  for (int k = 1; k < K; ++k) {
    scalar_t val = slice_start[k * inner_size];
    if (val < min_val) {
      min_val = val;
      min_index = k;
    }
  }

  // Write the result into the output tensor.
  output[outer * inner_size + inner] = min_index;
}

at::Tensor optimized_argmin_cuda_forward(const at::Tensor &x, int64_t dim) {
  // Ensure the input is a CUDA tensor.
  TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");

  int dims = x.dim();
  if (dim < 0) {
    dim += dims;
  }
  TORCH_CHECK(dim >= 0 && dim < dims, "Reduction dim out of range");

  int64_t outer_size = 1;
  for (int i = 0; i < dim; i++) {
    outer_size *= x.size(i);
  }
  int K = static_cast<int>(x.size(dim));
  int64_t inner_size = 1;
  for (int i = dim + 1; i < dims; i++) {
    inner_size *= x.size(i);
  }

  // Compute the output shape, which excludes the reduced dimension
  std::vector<int64_t> out_sizes;
  for (int i = 0; i < dims; i++) {
    if (i == dim) continue;
    out_sizes.push_back(x.size(i));
  }
  auto output = at::empty(out_sizes, x.options().dtype(at::kLong));

  // Transfer K to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(const_mem_k), &K, sizeof(int));

  // Configure the kernel launch
  int threads = 256;
  dim3 block_dim(threads);
  dim3 grid_dim((inner_size * outer_size + threads - 1) / threads);

  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, x.scalar_type(), "optimized_argmin_cuda_forward", ([&] {
    const scalar_t* x_data = x.data_ptr<scalar_t>();
    int64_t* output_data = output.data_ptr<int64_t>();
    optimized_argmin_kernel<scalar_t><<<grid_dim, block_dim>>>(x_data, output_data, inner_size);
  }));

  // Check for kernel launch errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    throw std::runtime_error(std::string("CUDA kernel failed: ") + hipGetErrorString(err));
  }

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &optimized_argmin_cuda_forward, "Optimized Argmin forward (CUDA)");
}