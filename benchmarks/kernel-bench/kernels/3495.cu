#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

template <typename scalar_t>
__device__ inline scalar_t gelu_function(scalar_t x);

template <>
__device__ inline float gelu_function<float>(float x) {
    return x * 0.5f * (1.0f + erff(x * 0.7071067811865475f));
}

template <>
__device__ inline double gelu_function<double>(double x) {
    return x * 0.5 * (1.0 + erf(x * 0.7071067811865475));
}

template <typename scalar_t, int VEC_SIZE>
__global__ void gelu_vectorized_kernel(const scalar_t* __restrict__ input,
                                      scalar_t* __restrict__ output,
                                      size_t n) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid + VEC_SIZE <= n) {
        #pragma unroll
        for (int i = 0; i < VEC_SIZE; ++i) {
            scalar_t val = __ldg(&input[tid + i]);  // Use texture cache
            output[tid + i] = gelu_function(val);
        }
    } else if (tid < n) {
        for (int i = 0; i < VEC_SIZE; ++i) {
            int index = tid + i;
            if (index < n) {
                scalar_t val = __ldg(&input[index]);  // Use texture cache
                output[index] = gelu_function(val);
            }
        }
    }
}

torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
    auto y = torch::empty_like(x);
    const size_t n = x.numel();
    
    constexpr int VEC_SIZE = 4;
    const int threads = 256;
    const int blocks = (n + threads * VEC_SIZE - 1) / (threads * VEC_SIZE);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "gelu_forward", [&] {
        gelu_vectorized_kernel<scalar_t, VEC_SIZE>
            <<<blocks, threads>>>(x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), n);
    });

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA error: ", hipGetErrorString(err));
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "GELU forward with optimized texture cache (CUDA)");
}