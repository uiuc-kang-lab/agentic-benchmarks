#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Adaptive block size based on hardware capabilities and kernel requirements
#define BLOCK_SIZE 128  // Experiment with different block sizes like 32, 64, 128, 256, 512
#define SMALL_MATRIX_DIM 128

// Macros for input checks
#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Static cuBLAS handle for fallback
static hipblasHandle_t cublas_handle = nullptr;

// Tiled matrix multiplication kernel with adaptive block size
__global__ void adaptive_blocksize_matmul_kernel(const float* __restrict__ A,
                                                  const float* __restrict__ B,
                                                  float* __restrict__ C,
                                                  const int M, const int N, const int K) {
    // Identify thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * BLOCK_SIZE + ty;
    int col = blockIdx.x * BLOCK_SIZE + tx;

    // Shared memory tiles for A and B
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    float sum = 0.0f;

    int numTiles = (K + BLOCK_SIZE - 1) / BLOCK_SIZE;
    for (int t = 0; t < numTiles; t++) {
        if (row < M && t * BLOCK_SIZE + tx < K)
            As[ty][tx] = A[row * K + t * BLOCK_SIZE + tx];
        else
            As[ty][tx] = 0.0f;

        if (t * BLOCK_SIZE + ty < K && col < N)
            Bs[ty][tx] = B[(t * BLOCK_SIZE + ty) * N + col];
        else
            Bs[ty][tx] = 0.0f;

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; k++) {
            sum += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

// Hybrid matrix multiplication: custom kernel for small matrices, cuBLAS for larger ones
void matrix_multiply_cuda(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);

    const int M = A.size(0);
    const int K = A.size(1);
    const int N = B.size(1);

    const float* d_A = A.data_ptr<float>();
    const float* d_B = B.data_ptr<float>();
    float* d_C = C.data_ptr<float>();

    // Use custom kernel for small matrices
    if (M <= SMALL_MATRIX_DIM && N <= SMALL_MATRIX_DIM && K <= SMALL_MATRIX_DIM) {
        dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
        adaptive_blocksize_matmul_kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);
        hipDeviceSynchronize();
    } else {
        // Use cuBLAS for larger matrices
        if (cublas_handle == nullptr) {
            hipblasCreate(&cublas_handle);
            hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
        }
        const float alpha = 1.0f;
        const float beta = 0.0f;
        // Note: cuBLAS assumes column-major order. When using row-major data, swap A and B.
        hipblasSgemm(cublas_handle,
                    HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N, M, K,
                    &alpha,
                    d_B, N,
                    d_A, K,
                    &beta,
                    d_C, N);
    }
}

// PyTorch forward interface
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    const int M = A.size(0);
    const int N = B.size(1);

    auto options = torch::TensorOptions()
                        .dtype(A.dtype())
                        .device(A.device())
                        .requires_grad(false);
    torch::Tensor C = torch::empty({M, N}, options);

    matrix_multiply_cuda(A, B, C);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Adaptive block size matrix multiplication (CUDA)");
}
