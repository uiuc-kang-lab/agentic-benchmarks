#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__device__ void load_tiles(const float* __restrict__ A, const float* __restrict__ B, 
                          float (&As)[TILE_SIZE][TILE_SIZE], float (&Bs)[TILE_SIZE][TILE_SIZE],
                          int row, int col, int tile, int N) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    if ((tile * TILE_SIZE + tx) <= row && row < N) {
        As[ty][tx] = A[row * N + (tile * TILE_SIZE + tx)];
    } else {
        As[ty][tx] = 0.0f;
    }

    if ((tile * TILE_SIZE + ty) < N && col < N) {
        Bs[ty][tx] = B[(tile * TILE_SIZE + ty) * N + col];
    } else {
        Bs[ty][tx] = 0.0f;
    }

    __syncthreads();
}

__device__ void compute_tile(float (&As)[TILE_SIZE][TILE_SIZE], 
                             float (&Bs)[TILE_SIZE][TILE_SIZE],
                             float &sum, int k_start, int k_end) {
    int tx = threadIdx.x;
    for (int k = k_start; k < k_end; ++k) {
        sum += As[threadIdx.y][k] * Bs[k][tx];
    }
    __syncthreads();
}

__global__ void modular_tril_mm_kernel(const float* __restrict__ A,
                                       const float* __restrict__ B,
                                       float* __restrict__ C,
                                       int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x * TILE_SIZE;
    int by = blockIdx.y * TILE_SIZE;
    int row = by + threadIdx.y;
    int col = bx + threadIdx.x;

    float sum = 0.0f;

    if (row >= col && row < N && col < N) {
        for (int tile = col / TILE_SIZE; tile <= row / TILE_SIZE; ++tile) {
            load_tiles(A, B, As, Bs, row, col, tile, N);
            int k_start = max(col, tile * TILE_SIZE);
            int k_end = min(row, (tile + 1) * TILE_SIZE);
            compute_tile(As, Bs, sum, k_start - tile * TILE_SIZE, k_end - tile * TILE_SIZE);
        }
        C[row * N + col] = sum;
    } else if (row < N && col < N) {
        C[row * N + col] = 0.0f;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    modular_tril_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Modular Triangular Matrix Multiplication (CUDA)");
}
