#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Combined CUDA kernel that leverages shared memory for weights and computes using unrolled loops
__global__ void conv1d_optimized_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int B,
    int in_channels,
    int in_size,
    int out_channels,
    int kernel_size,
    int out_size,
    int stride,
    int dilation
) {
    int b = blockIdx.x;
    int oc = blockIdx.y;

    // Allocate shared memory for the weight filter
    extern __shared__ float sweight[];
    int filter_size = in_channels * kernel_size;

    // Load the weight filter into shared memory
    for (int i = threadIdx.x; i < filter_size; i += blockDim.x) {
        sweight[i] = weight[oc * filter_size + i];
    }
    __syncthreads();

    float bias_val = (bias != nullptr) ? bias[oc] : 0.0f;

    // Each thread computes multiple output positions along the output width dimension
    for (int o = threadIdx.x; o < out_size; o += blockDim.x) {
        float sum = 0.0f;
        int start_pos = o * stride;
        int end_pos = start_pos + (kernel_size - 1) * dilation;

        if (end_pos < in_size) {
            // Avoid boundary checks if window is fully in bounds
            #pragma unroll
            for (int ic = 0; ic < in_channels; ++ic) {
                const float* x_ptr = x + b * (in_channels * in_size) + ic * in_size + start_pos;
                const float* w_ptr = sweight + ic * kernel_size;
                #pragma unroll
                for (int k = 0; k < kernel_size; ++k) {
                    sum += x_ptr[k * dilation] * w_ptr[k];
                }
            }
        } else {
            // In boundary cases, check each position
            #pragma unroll
            for (int ic = 0; ic < in_channels; ++ic) {
                const float* x_ptr = x + b * (in_channels * in_size) + ic * in_size;
                const float* w_ptr = sweight + ic * kernel_size;
                #pragma unroll
                for (int k = 0; k < kernel_size; ++k) {
                    int pos = start_pos + k * dilation;
                    if (pos < in_size) {
                        sum += x_ptr[pos] * w_ptr[k];
                    }
                }
            }
        }
        sum += bias_val;

        int out_index = b * (gridDim.y * out_size) + oc * out_size + o;
        output[out_index] = sum;
    }
}

// Forward function exposed via PyTorch
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int dilation) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(weight.dim() == 3, "weight must be 3D");
    TORCH_CHECK(weight.size(1) == x.size(1), "Input channels mismatch");

    if (bias.has_value()) {
        TORCH_CHECK(bias.value().device().is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias.value().is_contiguous(), "bias must be contiguous");
        TORCH_CHECK(bias.value().dim() == 1, "bias must be 1D");
        TORCH_CHECK(bias.value().size(0) == weight.size(0), "Bias size mismatch");
    }

    int B = x.size(0);
    int in_channels = x.size(1);
    int in_size = x.size(2);
    int out_channels = weight.size(0);
    int kernel_size = weight.size(2);

    int out_size = (in_size - dilation * (kernel_size - 1) - 1) / stride + 1;
    TORCH_CHECK(out_size > 0, "Invalid output size");

    auto output = torch::empty({B, out_channels, out_size}, x.options());
    if (output.numel() == 0) return output;

    const float* x_data = x.data_ptr<float>();
    const float* weight_data = weight.data_ptr<float>();
    const float* bias_data = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float* output_data = output.data_ptr<float>();

    dim3 blocks(B, out_channels);
    int threads = 256;
    int shared_mem_size = in_channels * kernel_size * sizeof(float);

    conv1d_optimized_kernel<<<blocks, threads, shared_mem_size>>>(
        x_data,
        weight_data,
        bias_data,
        output_data,
        B,
        in_channels,
        in_size,
        out_channels,
        kernel_size,
        out_size,
        stride,
        dilation);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel launch error: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized 1D convolution forward (CUDA)");
}
