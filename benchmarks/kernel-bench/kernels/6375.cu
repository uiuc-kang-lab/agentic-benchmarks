#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel that uses warp-level primitives to perform reduction over the specified dimension.

template <typename scalar_t>
__global__ void warp_sum_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t reduce_size,
    int64_t inner_size,
    int64_t total_output) {

    // Shared memory for tiling
    extern __shared__ scalar_t shared_mem[];
    
    // Each block handles one output element (one (outer, inner) pair).
    int idx = blockIdx.x;  // index for output element
    if (idx >= total_output) return;

    // Determine corresponding outer and inner indices
    int outer_idx = idx / inner_size;
    int inner_idx = idx % inner_size;

    scalar_t sum = 0;
    // Use lane id of the warp; assume blockDim.x == warpSize (32 threads)
    int lane = threadIdx.x;

    // Pre-calculate base offset to avoid redundant computations
    const int64_t base_offset = outer_idx * reduce_size * inner_size + inner_idx;
    
    // Process input in tiles to reduce global memory traffic
    const int TILE_SIZE = 128;  // Process 128 elements per tile
    
    // Each thread in the warp processes multiple elements across tiles
    for (int tile = 0; tile < reduce_size; tile += TILE_SIZE) {
        // Reset shared memory for this tile
        if (lane < TILE_SIZE) {
            shared_mem[lane] = 0;
        }
        __syncwarp();
        
        // Load tile data into shared memory
        #pragma unroll 4
        for (int i = lane; i < TILE_SIZE && (tile + i) < reduce_size; i += warpSize) {
            int64_t offset = base_offset + (tile + i) * inner_size;
            shared_mem[i] = input[offset];
        }
        __syncwarp();
        
        // Reduce within the tile
        #pragma unroll 4
        for (int i = 0; i < min(TILE_SIZE, reduce_size - tile); i++) {
            sum += shared_mem[i];
        }
        __syncwarp();
    }

    // Use warp-level shuffle to reduce the partial sums within the warp
    scalar_t partial = sum;
    const unsigned int mask = 0xffffffff;
    
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        const scalar_t n = __shfl_down_sync(mask, partial, offset);
        partial += n;
    }
    sum = partial;

    // The first lane writes the result
    if (lane == 0) {
        output[idx] = sum;
    }
}

// Host function wrapping the kernel launch

torch::Tensor sum_reduce_cuda(torch::Tensor input, int64_t dim) {
    // Adjust negative dimensions
    if (dim < 0) dim += input.dim();
    auto sizes = input.sizes().vec();
    int64_t reduce_size = sizes[dim];

    // Compute outer_size: product of dimensions before the reduction dim
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }

    // Compute inner_size: product of dimensions after the reduction dim
    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }

    // Prepare output tensor with reduce dimension set to 1
    sizes[dim] = 1;
    auto output = torch::empty(sizes, input.options());

    // Total number of output elements
    int64_t total_output = outer_size * inner_size;

    // Launch one warp (32 threads) per output element
    const int threads = 32;  // warp size
    const int blocks = total_output;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sum_reduce_cuda", ([&] {
        warp_sum_reduce_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            reduce_size,
            inner_size,
            total_output
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &sum_reduce_cuda, "Sum reduction forward (CUDA) using warp-level primitives");
}
