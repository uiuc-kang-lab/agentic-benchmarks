#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void depthwise_conv2d_kernel(
    const float* input,
    const float* weight,
    const float* bias,
    float* output,
    int batch_size,
    int in_channels,
    int input_h,
    int input_w,
    int out_channels,
    int output_h,
    int output_w,
    int kernel_size,
    int stride,
    int padding,
    int channels_per_group
) {
    int total_elements = batch_size * out_channels * output_h * output_w;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_elements) return;

    int w_out = idx % output_w;
    idx /= output_w;
    int h_out = idx % output_h;
    idx /= output_h;
    int oc = idx % out_channels;
    int b = idx / out_channels;

    int in_ch = oc / channels_per_group;
    int weight_ch = oc % channels_per_group;

    float sum = 0.0f;
    int h_in_start = h_out * stride - padding;
    int w_in_start = w_out * stride - padding;
    
    // Use warp-level primitives for partial sums
    const unsigned FULL_MASK = 0xffffffff;
    int lane_id = threadIdx.x % 32;
    float warp_sum = 0.0f;

    for (int kh = 0; kh < kernel_size; ++kh) {
        int h_in = h_in_start + kh;
        if (h_in >= 0 && h_in < input_h) {
            for (int kw = 0; kw < kernel_size; ++kw) {
                int w_in = w_in_start + kw;
                if (w_in >= 0 && w_in < input_w) {
                    int input_idx = b * (in_channels * input_h * input_w)
                                  + in_ch * (input_h * input_w)
                                  + h_in * input_w
                                  + w_in;
                                  
                    int weight_idx = in_ch * (channels_per_group * kernel_size * kernel_size)
                                   + weight_ch * (kernel_size * kernel_size)
                                   + kh * kernel_size
                                   + kw;

                    warp_sum += input[input_idx] * weight[weight_idx];
                }
            }
        }
    }

    // Perform warp-level reduction
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2) {
        warp_sum += __shfl_down_sync(FULL_MASK, warp_sum, offset);
    }

    // First thread in each warp has the final sum
    if (lane_id == 0) {
        sum = warp_sum;
    }
    
    // Broadcast the result to all threads in the warp
    sum = __shfl_sync(FULL_MASK, sum, 0);

    if (bias != nullptr) {
        sum += bias[oc];
    }
    
    if (idx < total_elements) {
        output[b * out_channels * output_h * output_w +
               oc * output_h * output_w +
               h_out * output_w +
               w_out] = sum;
    }
}

torch::Tensor forward(
    torch::Tensor input,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding
) {
    TORCH_CHECK(input.is_cuda() && weight.is_cuda(), "Inputs must be CUDA tensors");
    if (bias.has_value()) {
        TORCH_CHECK(bias->is_cuda(), "Bias must be a CUDA tensor");
    }
    TORCH_CHECK(input.is_contiguous() && weight.is_contiguous(), "Input and weight must be contiguous");
    if (bias.has_value()) {
        TORCH_CHECK(bias->is_contiguous(), "Bias must be contiguous");
    }
    TORCH_CHECK(weight.dim() == 4, "Weight must be 4D tensor");

    int batch_size = input.size(0);
    int in_channels = input.size(1);
    int input_h = input.size(2);
    int input_w = input.size(3);
    int kernel_size = weight.size(2);
    int channels_per_group = weight.size(1);
    int out_channels = in_channels * channels_per_group;

    if (bias.has_value()) {
        TORCH_CHECK(bias->size(0) == out_channels, "Bias size mismatch");
    }

    int output_h = (input_h + 2 * padding - kernel_size) / stride + 1;
    int output_w = (input_w + 2 * padding - kernel_size) / stride + 1;

    auto output = torch::empty({batch_size, out_channels, output_h, output_w}, input.options());

    int total_elements = batch_size * out_channels * output_h * output_w;
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;

    const float* bias_ptr = bias ? bias->data_ptr<float>() : nullptr;

    depthwise_conv2d_kernel<<<blocks, threads>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        input_h,
        input_w,
        out_channels,
        output_h,
        output_w,
        kernel_size,
        stride,
        padding,
        channels_per_group
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Depthwise 2D Convolution (CUDA)",
          py::arg("input"), py::arg("weight"), py::arg("bias") = py::none(), py::arg("stride"), py::arg("padding"));
}