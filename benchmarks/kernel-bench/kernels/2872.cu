#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel function to compute sigmoid
template <typename scalar_t>
__global__ void sigmoid_kernel(const scalar_t* __restrict__ input,
                               scalar_t* __restrict__ output,
                               const int64_t size) {
  const int stride = blockDim.x * gridDim.x * blockDim.y;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += stride) {
    float val = static_cast<float>(-input[i]);
    float exp_val = expf(val);
    float r = 1.0f / (1.0f + exp_val);
    output[i] = static_cast<scalar_t>(r);
  }
}

// Forward function with CUDA streams
torch::Tensor forward(torch::Tensor input) {
  auto output = torch::empty_like(input);
  const int64_t size = input.numel();

  const int threads = 256;
  const int max_blocks = 65535;  // Maximum blocks per grid dimension
  const int min_blocks = (size + threads - 1) / threads;
  const int blocks = min(max_blocks, min_blocks);

  // Create CUDA stream
  hipStream_t stream;
  hipStreamCreate(&stream);

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sigmoid_kernel", [&] {
    const auto* input_data = input.data_ptr<scalar_t>();
    auto* output_data = output.data_ptr<scalar_t>();

    // Launch kernel on the created stream
    sigmoid_kernel<scalar_t><<<blocks, threads, 0, stream>>>(input_data, output_data, size);
  });

  // Synchronize the stream
  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "Sigmoid forward (CUDA) with streams");
}