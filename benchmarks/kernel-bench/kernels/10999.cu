#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static const int BLOCK_SIZE = 128;

template <typename scalar_t>
__global__ void mse_forward_kernel(
    const scalar_t* __restrict__ preds,
    const scalar_t* __restrict__ tgts,
    double* __restrict__ sum_out,
    const int64_t num_elements
) {
    __shared__ double shm[BLOCK_SIZE];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double thread_sum = 0.0;

    // Optimized strided loop with sequential memory access
    while (idx < num_elements) {
        double diff = static_cast<double>(preds[idx]) - static_cast<double>(tgts[idx]);
        thread_sum += diff * diff;
        idx += blockDim.x * gridDim.x;
    }

    shm[threadIdx.x] = thread_sum;
    __syncthreads();

    // Enhanced reduction with better memory access pattern
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shm[threadIdx.x] += shm[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicAdd(sum_out, shm[0]);
    }
}

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.is_cuda(), "predictions must be CUDA tensor");
    TORCH_CHECK(targets.is_cuda(), "targets must be CUDA tensor");
    TORCH_CHECK(predictions.numel() == targets.numel(), "Input sizes must match");

    const int64_t num_elements = predictions.numel();
    auto accumulator = torch::zeros({1}, predictions.options().dtype(at::kDouble));

    // Balanced grid size calculation for 128-block
    const int grid_size = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    AT_DISPATCH_FLOATING_TYPES(predictions.scalar_type(), "mse_forward", [&] {
        mse_forward_kernel<scalar_t><<<grid_size, BLOCK_SIZE>>>(
            predictions.data_ptr<scalar_t>(),
            targets.data_ptr<scalar_t>(),
            accumulator.data_ptr<double>(),
            num_elements
        );
    });

    return accumulator.div_(num_elements).to(predictions.dtype());
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "MSE forward (CUDA optimized block size");
}