#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__constant__ int d_num_features;
__constant__ float d_eps;

template <typename scalar_t>
__device__ scalar_t calculate_sumsq(
    const scalar_t* __restrict__ input,
    const int batch_offset,
    const int numel_per_batch,
    const int offset_in_batch
) {
    scalar_t sumsq = 0.0f;
    for (int feat = 0; feat < d_num_features; feat++) {
        const scalar_t val = input[batch_offset + feat * numel_per_batch + offset_in_batch];
        sumsq += val * val;
    }
    return sumsq;
}

template <typename scalar_t>
__device__ void normalize_features(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_offset,
    const int numel_per_batch,
    const int offset_in_batch,
    const scalar_t rms
) {
    const scalar_t inv_rms = static_cast<scalar_t>(1.0) / rms;
    for (int feat = 0; feat < d_num_features; feat++) {
        const int idx = batch_offset + feat * numel_per_batch + offset_in_batch;
        output[idx] = input[idx] * inv_rms;
    }
}

template <typename scalar_t>
__global__ void rms_norm_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int numel_per_batch
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int batch_id = tid / numel_per_batch;
    
    if (batch_id >= batch_size) return;
    
    const int offset_in_batch = tid % numel_per_batch;
    const int batch_offset = batch_id * d_num_features * numel_per_batch;

    // Calculate sum of squares using modular function
    const scalar_t sumsq = calculate_sumsq(
        input, batch_offset, numel_per_batch, 
        offset_in_batch
    );
    
    // Calculate RMS
    const scalar_t rms = sqrt(sumsq / d_num_features + d_eps);
    
    // Normalize using modular function
    normalize_features(
        input, output, batch_offset, numel_per_batch,
        offset_in_batch, rms
    );
}

torch::Tensor rms_norm_cuda_forward(torch::Tensor input, float eps) {
    auto output = torch::empty_like(input);
    
    const int batch_size = input.size(0);
    const int num_features = input.size(1);
    
    int numel_per_batch = 1;
    for(int i = 2; i < input.dim(); i++) {
        numel_per_batch *= input.size(i);
    }

    // Copy constants to device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_features), &num_features, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_eps), &eps, sizeof(float));

    const int total_threads = batch_size * numel_per_batch;
    const int threads_per_block = 256;
    const int blocks = (total_threads + threads_per_block - 1) / threads_per_block;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "rms_norm_cuda", ([&] {
        rms_norm_kernel<scalar_t><<<blocks, threads_per_block>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            numel_per_batch
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &rms_norm_cuda_forward, "RMS normalization forward (CUDA)");
}