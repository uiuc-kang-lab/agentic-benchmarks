#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <c10/util/Optional.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

__global__ void initialize_output_kernel(
    float* __restrict__ output,
    const float* __restrict__ bias,
    const int batch,
    const int out_channels,
    const int out_h,
    const int out_w) {
    
    const int total = batch * out_channels * out_h * out_w;
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < total; idx += blockDim.x * gridDim.x) {
        const int oc = (idx / out_w / out_h) % out_channels;
        output[idx] = __ldg(&bias[oc]);
    }
}

__global__ void conv_transposed2d_scatter_atomic_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    float* __restrict__ output,
    const int batch,
    const int in_channels,
    const int in_h,
    const int in_w,
    const int out_channels_per_group,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_h,
    const int pad_w,
    const int dilation_h,
    const int dilation_w,
    const int groups,
    const int out_h,
    const int out_w,
    const int in_channels_per_group) {

    const int total = batch * in_channels * in_h * in_w;
    const int elements_per_thread = 4;
    
    for (int linear_idx = blockIdx.x * blockDim.x * elements_per_thread + threadIdx.x;
         linear_idx < total;
         linear_idx += blockDim.x * gridDim.x * elements_per_thread) {
        
        #pragma unroll
        for (int i = 0; i < elements_per_thread; ++i) {
            const int index = linear_idx + i * blockDim.x * gridDim.x;
            if (index >= total) break;

            const int iw = index % in_w;
            const int tmp1 = index / in_w;
            const int ih = tmp1 % in_h;
            const int tmp2 = tmp1 / in_h;
            const int c = tmp2 % in_channels;
            const int n = tmp2 / in_channels;

            const float x_val = __ldg(&x[index]);
            const int group = c / in_channels_per_group;
            const int weight_base = c * (out_channels_per_group * kernel_h * kernel_w);
            const int output_batch_offset = n * (groups * out_channels_per_group * out_h * out_w);
            const int group_offset = group * out_channels_per_group;

            #pragma unroll 4
            for (int kh = 0; kh < kernel_h; ++kh) {
                const int out_row = ih * stride_h - pad_h + kh * dilation_h;
                if (out_row < 0 || out_row >= out_h) continue;

                #pragma unroll 4
                for (int kw = 0; kw < kernel_w; ++kw) {
                    const int out_col = iw * stride_w - pad_w + kw * dilation_w;
                    if (out_col < 0 || out_col >= out_w) continue;

                    const int kernel_offset = (kernel_h - 1 - kh) * kernel_w + (kernel_w - 1 - kw);
                    const int weight_start = weight_base + kernel_offset;

                    #pragma unroll 4
                    for (int oc_offset = 0; oc_offset < out_channels_per_group; ++oc_offset) {
                        const float weight_val = __ldg(&weight[weight_start + oc_offset * kernel_h * kernel_w]);
                        const int out_index = output_batch_offset + 
                                            (group_offset + oc_offset) * (out_h * out_w) + 
                                            out_row * out_w + out_col;
                        atomicAdd(&output[out_index], x_val * weight_val);
                    }
                }
            }
        }
    }
}

at::Tensor forward(
    at::Tensor x,
    at::Tensor weight,
    c10::optional<at::Tensor> bias,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> dilation,
    int groups) {
    
    x = x.contiguous();
    weight = weight.contiguous();
    
    if (!bias.has_value() || !bias.value().defined()) {
        bias = at::zeros({weight.size(1) * groups}, weight.options());
    } else {
        bias = bias.value().contiguous();
    }

    const int batch = x.size(0);
    const int in_channels = x.size(1);
    const int in_h = x.size(2);
    const int in_w = x.size(3);
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);
    const int out_channels_per_group = weight.size(1);
    const int out_channels = out_channels_per_group * groups;
    const int in_channels_per_group = in_channels / groups;
    
    const int stride_h = stride[0];
    const int stride_w = stride[1];
    const int pad_h = padding[0];
    const int pad_w = padding[1];
    const int dilation_h = dilation[0];
    const int dilation_w = dilation[1];

    const int out_h = (in_h - 1) * stride_h - 2 * pad_h + dilation_h * (kernel_h - 1) + 1;
    const int out_w = (in_w - 1) * stride_w - 2 * pad_w + dilation_w * (kernel_w - 1) + 1;

    auto output = at::empty({batch, out_channels, out_h, out_w}, x.options());

    // Initialize output with bias
    const int total_output = batch * out_channels * out_h * out_w;
    const int threads_init = 512;
    const int blocks_init = (total_output + threads_init - 1) / threads_init;
    initialize_output_kernel<<<blocks_init, threads_init>>>(
        output.data_ptr<float>(),
        bias.value().data_ptr<float>(),
        batch,
        out_channels,
        out_h,
        out_w);

    // Scatter kernel with grid-stride loops
    const int total_input = batch * in_channels * in_h * in_w;
    const int threads_scatter = 512;
    const int blocks_scatter = min(65535, (total_input + threads_scatter * 4 - 1) / (threads_scatter * 4));
    
    conv_transposed2d_scatter_atomic_kernel<<<blocks_scatter, threads_scatter>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        output.data_ptr<float>(),
        batch,
        in_channels,
        in_h,
        in_w,
        out_channels_per_group,
        kernel_h,
        kernel_w,
        stride_h,
        stride_w,
        pad_h,
        pad_w,
        dilation_h,
        dilation_w,
        groups,
        out_h,
        out_w,
        in_channels_per_group);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel failed: %s\n", hipGetErrorString(err));
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "2D Transposed Convolution with Optimized Grid-Stride (CUDA)",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride"),
          py::arg("padding"),
          py::arg("dilation"),
          py::arg("groups"));
}