#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void max_pool2d_balanced_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int ow = blockIdx.x * blockDim.x + threadIdx.x;
    const int oh = blockIdx.y * blockDim.y + threadIdx.y;
    const int c = blockIdx.z % channels;
    const int b = blockIdx.z / channels;

    if (ow >= output_width || oh >= output_height || b >= batch_size) return;

    const int input_batch_stride = channels * input_height * input_width;
    const int input_channel_stride = input_height * input_width;
    const int base_idx = b * input_batch_stride + c * input_channel_stride;
    
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    const int ih_start = oh * stride - padding;
    const int iw_start = ow * stride - padding;

    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = ih_start + kh * dilation;
        if (ih >= 0 && ih < input_height) {
            const int row_idx = base_idx + ih * input_width;
            for (int kw = 0; kw < kernel_size; kw++) {
                const int iw = iw_start + kw * dilation;
                if (iw >= 0 && iw < input_width) {
                    max_val = max(max_val, __ldg(&input[row_idx + iw]));
                }
            }
        }
    }

    const int output_idx = b * (channels * output_height * output_width) +
                           c * (output_height * output_width) +
                           oh * output_width +
                           ow;
    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const dim3 threads(16, 16);
    const dim3 blocks((output_width + threads.x - 1) / threads.x,
                      (output_height + threads.y - 1) / threads.y,
                      batch_size * channels);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_balanced_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size, channels,
            input_height, input_width,
            output_height, output_width,
            kernel_size, stride, padding, dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}