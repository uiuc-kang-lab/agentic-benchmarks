#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void triplet_margin_loss_kernel_warp_optimized(
    const scalar_t* anchor,
    const scalar_t* positive,
    const scalar_t* negative,
    scalar_t* output,
    const float margin,
    const int batch_size,
    const int feat_size) {
    
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int batch_idx = tid / feat_size;
    const int feat_idx = tid % feat_size;
    
    if (batch_idx < batch_size) {
        const int idx = batch_idx * feat_size + feat_idx;
        const scalar_t a = anchor[idx];
        const scalar_t p = positive[idx];
        const scalar_t n = negative[idx];
        
        // Compute distance components
        const scalar_t d_pos = a - p;
        const scalar_t d_neg = a - n;
        
        // Squared distances
        scalar_t dist_pos = d_pos * d_pos;
        scalar_t dist_neg = d_neg * d_neg;
        
        // Use shared memory for reduction
        extern __shared__ scalar_t shared_mem[];
        shared_mem[threadIdx.x] = dist_pos - dist_neg;
        __syncthreads();
        
        // Reduce within feature dimension
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                shared_mem[threadIdx.x] += shared_mem[threadIdx.x + stride];
            }
            __syncthreads();
        }
        
        if (threadIdx.x == 0) {
            // Final loss computation for this batch element
            const scalar_t loss = max(scalar_t(0.0), sqrt(shared_mem[0]) + margin);
            atomicAdd(&output[batch_idx], loss);
        }
    }
}

torch::Tensor triplet_margin_loss_cuda_warp_optimized(
    torch::Tensor anchor,
    torch::Tensor positive,
    torch::Tensor negative,
    float margin) {
    
    TORCH_CHECK(anchor.device().is_cuda(), "anchor must be a CUDA tensor");
    TORCH_CHECK(positive.device().is_cuda(), "positive must be a CUDA tensor");
    TORCH_CHECK(negative.device().is_cuda(), "negative must be a CUDA tensor");
    
    const int batch_size = anchor.size(0);
    const int feat_size = anchor.size(1);
    auto output = torch::zeros({batch_size}, anchor.options());
    
    const int threads = 256;
    const int blocks = (batch_size * feat_size + threads - 1) / threads;
    const int shared_mem_size = threads * sizeof(scalar_t);
    
    AT_DISPATCH_FLOATING_TYPES(anchor.scalar_type(), "triplet_margin_loss_kernel_warp_optimized", ([&] {
        triplet_margin_loss_kernel_warp_optimized<scalar_t><<<blocks, threads, shared_mem_size>>>(
            anchor.data_ptr<scalar_t>(),
            positive.data_ptr<scalar_t>(),
            negative.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            margin,
            batch_size,
            feat_size);
    }));
    
    return output.mean();
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &triplet_margin_loss_cuda_warp_optimized, "Triplet margin loss forward (CUDA)");
}