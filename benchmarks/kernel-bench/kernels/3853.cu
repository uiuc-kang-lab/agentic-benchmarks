#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <type_traits>

// Numerically stable softplus function
template <typename scalar_t>
__device__ __forceinline__ scalar_t softplus_fn(scalar_t x) {
    if (x > static_cast<scalar_t>(20.0)) {
        return x;
    } else if (x < static_cast<scalar_t>(-20.0)) {
        return exp(x);
    } else {
        return log1p(exp(x));
    }
}

// Traits to define vectorized types for coalesced memory access
template <typename T>
struct vectorized_traits;

// For float, process 4 elements at a time using float4 (16 bytes)
template <>
struct vectorized_traits<float> {
    using VecType = float4;
    static constexpr int vec_size = 4;
};

// For double, process 2 elements at a time using double2 (16 bytes)
template <>
struct vectorized_traits<double> {
    using VecType = double2;
    static constexpr int vec_size = 2;
};

// Kernel that processes the bulk of the tensor using vectorized loads/stores
// Ensures memory coalescing by having consecutive threads load contiguous memory
template <typename scalar_t>
__global__ void softplus_kernel_vectorized(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int n_vec) {

    using VecType = typename vectorized_traits<scalar_t>::VecType;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Reinterpret pointers as vectorized pointers
    const VecType* __restrict__ input_vec = reinterpret_cast<const VecType*>(input);
    VecType* __restrict__ output_vec = reinterpret_cast<VecType*>(output);

    for (int i = idx; i < n_vec; i += stride) {
        VecType vec = input_vec[i];
        if constexpr (std::is_same<scalar_t, float>::value) {
            vec.x = softplus_fn(vec.x);
            vec.y = softplus_fn(vec.y);
            vec.z = softplus_fn(vec.z);
            vec.w = softplus_fn(vec.w);
        } else { // double
            vec.x = softplus_fn(vec.x);
            vec.y = softplus_fn(vec.y);
        }
        output_vec[i] = vec;
    }
}

// Scalar kernel to process any remaining elements (in case total size is not a multiple of vec_size)
template <typename scalar_t>
__global__ void softplus_kernel_scalar(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int start,
    const int size) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = start + idx; i < size; i += stride) {
        output[i] = softplus_fn(input[i]);
    }
}

// CUDA forward function
torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "softplus_forward_cuda", ([&] {
        // Determine vector size based on the scalar type
        const int vec_size = vectorized_traits<scalar_t>::vec_size;
        int n_vec = size / vec_size;
        int remainder = size - n_vec * vec_size;

        const int threads = 256;
        int blocks = (n_vec + threads - 1) / threads;

        // Launch vectorized kernel if there are full vector packs
        if (n_vec > 0) {
            softplus_kernel_vectorized<scalar_t><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                n_vec);
        }
        // Process any remaining elements with a scalar kernel
        if (remainder > 0) {
            int start = n_vec * vec_size;
            int blocks_scalar = (remainder + threads - 1) / threads;
            softplus_kernel_scalar<scalar_t><<<blocks_scalar, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                start,
                size);
        }
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}
