#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized warp reduction using shuffle instructions
template <typename scalar_t>
__device__ __forceinline__ scalar_t warp_reduce_sum(scalar_t val) {
    unsigned int mask = 0xffffffff;
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(mask, val, offset);
    }
    return val;
}

// Hybrid block reduction combining shared memory and warp shuffles
template <typename scalar_t>
__device__ __forceinline__ scalar_t hybrid_block_reduce(
    scalar_t* sdata,
    scalar_t thread_sum,
    const int tid,
    const int blockSize) {
    
    const int lane = tid & 31;
    const int wid = tid >> 5;
    const int warps = blockSize >> 5;
    
    // First do warp-level reduction
    scalar_t warp_sum = warp_reduce_sum(thread_sum);
    
    // Write reduced warp sums to shared memory
    if (lane == 0) {
        sdata[wid] = warp_sum;
    }
    __syncthreads();
    
    // Final reduction: let first warp handle all partial sums
    scalar_t block_sum = 0;
    if (tid < warps) {
        block_sum = warp_reduce_sum(tid < warps ? sdata[tid] : 0);
    }
    return block_sum;
}

template <typename scalar_t>
__global__ void hybrid_mean_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t outer_size,
    int64_t dim_size,
    int64_t inner_size) {
    
    extern __shared__ char shared_mem[];
    scalar_t* sdata = reinterpret_cast<scalar_t*>(shared_mem);
    
    const int tid = threadIdx.x;
    const int out_idx = blockIdx.x;
    
    if (out_idx >= outer_size * inner_size)
        return;
        
    const int outer_idx = out_idx / inner_size;
    const int inner_idx = out_idx % inner_size;
    const int base_idx = outer_idx * dim_size * inner_size + inner_idx;
    
    // Coalesced memory access pattern with grid-stride loop
    scalar_t thread_sum = 0;
    #pragma unroll
    for (int i = tid; i < dim_size; i += blockDim.x) {
        thread_sum += input[base_idx + i * inner_size];
    }
    
    // Perform hybrid reduction
    scalar_t block_sum = hybrid_block_reduce(sdata, thread_sum, tid, blockDim.x);
    
    if (tid == 0) {
        output[out_idx] = block_sum / static_cast<scalar_t>(dim_size);
    }
}

torch::Tensor mean_reduce_cuda(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();
    
    auto sizes = input.sizes().vec();
    int64_t dim_size = sizes[dim];
    
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }
    
    int64_t inner_size = 1;
    for (size_t i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }
    
    sizes.erase(sizes.begin() + dim);
    auto output = torch::empty(sizes, input.options());
    
    const int threads = 256;
    const int blocks = outer_size * inner_size;
    const int warps_per_block = threads >> 5;
    
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "hybrid_mean_reduce", ([&] {
        const int shared_mem_size = warps_per_block * sizeof(scalar_t);
        hybrid_mean_reduce_kernel<scalar_t><<<blocks, threads, shared_mem_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            dim_size,
            inner_size
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &mean_reduce_cuda, "Hybrid mean reduction (CUDA)");
}