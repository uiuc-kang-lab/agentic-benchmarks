#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32  // Increased tile size for better occupancy
#define WARP_SIZE 32

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   const int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    // Early exit if above diagonal
    if (row < col || row >= N || col >= N) {
        return;
    }

    float sum = 0.0f;
    
    // Calculate number of tiles needed
    const int numTiles = (min(row, N-1) - col + TILE_SIZE) / TILE_SIZE;
    
    #pragma unroll 2
    for (int t = 0; t < numTiles; t++) {
        const int tileStart = col + t * TILE_SIZE;
        
        // Collaborative loading with vectorized reads where possible
        if (tileStart + tx <= row && row < N) {
            As[ty][tx] = A[row * N + (tileStart + tx)];
        } else {
            As[ty][tx] = 0.0f;
        }
        
        if (tileStart + ty < N && col < N) {
            Bs[ty][tx] = B[(tileStart + ty) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute partial sum for this tile using warp-level optimizations
        if (row < N && col < N) {
            #pragma unroll 8
            for (int k = 0; k < TILE_SIZE; k++) {
                if ((tileStart + k) >= col && (tileStart + k) <= row) {
                    sum = __fmaf_rn(As[ty][k], Bs[k][tx], sum);
                }
            }
        }
        
        __syncthreads();
    }
    
    // Write result
    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D tensors");
    TORCH_CHECK(A.size(0) == A.size(1) && B.size(0) == B.size(1), "A and B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    const int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE, 
                   (N + TILE_SIZE - 1) / TILE_SIZE);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    triangular_mm_kernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized triangular matrix multiplication (CUDA)");
}