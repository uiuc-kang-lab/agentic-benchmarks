#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized CUDA kernel for batched matrix multiplication using stride loops
// A: (batch_size, M, K), B: (batch_size, K, N), C: (batch_size, M, N)
__global__ void bmm_kernel_stride(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int batch_size,
    int M,
    int K,
    int N
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int total = batch_size * M * N;

    // Stride loop to cover all output elements
    for (int index = idx; index < total; index += stride) {
        int b = index / (M * N);
        int rem = index % (M * N);
        int m = rem / N;
        int n = rem % N;

        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += A[b * M * K + m * K + k] * B[b * K * N + k * N + n];
        }
        C[b * M * N + m * N + n] = sum;
    }
}

// Torch binding function
torch::Tensor forward_bmm(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 3, "A must be 3D");
    TORCH_CHECK(B.dim() == 3, "B must be 3D");
    TORCH_CHECK(A.size(0) == B.size(0), "Batch sizes must match");
    TORCH_CHECK(A.size(2) == B.size(1), "Inner dimensions (K) must match");

    int batch_size = A.size(0);
    int M = A.size(1);
    int K = A.size(2);
    int N = B.size(2);

    auto options = torch::TensorOptions().dtype(A.dtype()).device(A.device());
    auto C = torch::zeros({batch_size, M, N}, options);

    int total = batch_size * M * N;
    const int threads = 256;
    int blocks = (total + threads - 1) / threads;

    bmm_kernel_stride<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        batch_size, M, K, N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_bmm, "Batched matrix multiplication (CUDA) with stride loops");
}
