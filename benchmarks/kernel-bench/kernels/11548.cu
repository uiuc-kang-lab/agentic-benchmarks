#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ __forceinline__ float compute_kl_div(float log_pred, float target) {
    return __expf(log_pred) - target * log_pred;
}

__device__ __forceinline__ float warp_reduce_sum(float val) {
    #pragma unroll
    for (int offset = 32/2; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__global__ void balanced_workload_kl_div_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int num_threads = blockDim.x;
    const int warp_id = tid / 32;
    const int lane_id = tid % 32;
    const int global_id = bid * num_threads + tid;
    
    // Local accumulator
    float thread_sum = 0.0f;
    
    // Distribute workload evenly using grid-stride loop
    for (int i = global_id; i < n; i += gridDim.x * num_threads) {
        thread_sum += compute_kl_div(log_predictions[i], targets[i]);
    }
    
    // Warp-level reduction
    thread_sum = warp_reduce_sum(thread_sum);
    
    // Shared memory for warp results
    __shared__ float warp_sums[32];
    if (lane_id == 0) {
        warp_sums[warp_id] = thread_sum;
    }
    __syncthreads();
    
    // Final reduction by first warp
    if (warp_id == 0) {
        float sum = (tid < blockDim.x / 32) ? warp_sums[lane_id] : 0.0f;
        sum = warp_reduce_sum(sum);
        
        if (lane_id == 0) {
            atomicAdd(output, sum);
        }
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    // Launch parameters
    const int threads = 256; // Multiple of warp size
    const int blocks = min((n + threads - 1) / threads, 1024);
    
    balanced_workload_kl_div_kernel<<<blocks, threads>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward (CUDA)");
}