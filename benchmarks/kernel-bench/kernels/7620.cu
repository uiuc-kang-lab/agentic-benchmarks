#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Constant memory declarations for frequently accessed parameters
__constant__ int c_stride[3];
__constant__ int c_padding[3];
__constant__ int c_dims[10];  // Store N, C_in, D_in, H_in, W_in, C_out, D_out, H_out, W_out, groups

// Optimized CUDA kernel using constant memory
__global__ void conv_transposed_3d_cuda_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int totalElements,
    int kD, int kH, int kW
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    while (idx < totalElements) {
        // Decode flat index using constant memory dimensions
        int w = idx % c_dims[8];  // W_out
        int tmp = idx / c_dims[8];
        int h = tmp % c_dims[7];  // H_out
        tmp /= c_dims[7];
        int d = tmp % c_dims[6];  // D_out
        tmp /= c_dims[6];
        int c_out = tmp % c_dims[5];  // C_out
        int n = tmp / c_dims[5];

        // Group calculations using constant memory
        int output_channels_per_group = c_dims[5] / c_dims[9];  // C_out / groups
        int group = c_out / output_channels_per_group;
        int c_out_in_group = c_out - group * output_channels_per_group;
        int input_channels_per_group = c_dims[1] / c_dims[9];  // C_in / groups

        // Initialize accumulator
        float out_val = (bias != nullptr) ? bias[c_out] : 0.0f;

        for (int r = 0; r < kD; r++) {
            int d_in_calc = d + c_padding[0] - r;
            if (d_in_calc % c_stride[0] != 0) continue;
            int d_in = d_in_calc / c_stride[0];
            if (d_in < 0 || d_in >= c_dims[2]) continue;
            
            for (int s = 0; s < kH; s++) {
                int h_in_calc = h + c_padding[1] - s;
                if (h_in_calc % c_stride[1] != 0) continue;
                int h_in = h_in_calc / c_stride[1];
                if (h_in < 0 || h_in >= c_dims[3]) continue;
                
                for (int t = 0; t < kW; t++) {
                    int w_in_calc = w + c_padding[2] - t;
                    if (w_in_calc % c_stride[2] != 0) continue;
                    int w_in = w_in_calc / c_stride[2];
                    if (w_in < 0 || w_in >= c_dims[4]) continue;

                    for (int c = 0; c < input_channels_per_group; c++) {
                        int actual_c_in = group * input_channels_per_group + c;
                        
                        // Calculate indices using constant memory dimensions
                        int input_index = (((n * c_dims[1] + actual_c_in) * c_dims[2] + d_in) 
                                         * c_dims[3] + h_in) * c_dims[4] + w_in;
                        float in_val = input[input_index];
                        
                        int weight_index = ((actual_c_in * output_channels_per_group + c_out_in_group) 
                                          * (kD * kH * kW)) + (r * kH * kW + s * kW + t);
                        float w_val = weight[weight_index];
                        
                        out_val += in_val * w_val;
                    }
                }
            }
        }
        
        output[idx] = out_val;
        idx += blockDim.x * gridDim.x;
    }
}

torch::Tensor forward(
    torch::Tensor input,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> output_padding,
    int64_t groups
) {
    // Get dimensions
    const int N = input.size(0);
    const int C_in = input.size(1);
    const int D_in = input.size(2);
    const int H_in = input.size(3);
    const int W_in = input.size(4);
    const int kD = weight.size(2);
    const int kH = weight.size(3);
    const int kW = weight.size(4);
    
    // Calculate output dimensions
    const int output_channels_per_group = weight.size(1);
    const int C_out = output_channels_per_group * groups;
    const int D_out = (D_in - 1) * stride[0] - 2 * padding[0] + kD + output_padding[0];
    const int H_out = (H_in - 1) * stride[1] - 2 * padding[1] + kH + output_padding[1];
    const int W_out = (W_in - 1) * stride[2] - 2 * padding[2] + kW + output_padding[2];

    // Copy constant parameters to device
    int h_stride[3] = {static_cast<int>(stride[0]), static_cast<int>(stride[1]), static_cast<int>(stride[2])};
    int h_padding[3] = {static_cast<int>(padding[0]), static_cast<int>(padding[1]), static_cast<int>(padding[2])};
    int h_dims[10] = {N, C_in, D_in, H_in, W_in, C_out, D_out, H_out, W_out, static_cast<int>(groups)};
    
    hipMemcpyToSymbol(HIP_SYMBOL(c_stride), h_stride, sizeof(int) * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(c_padding), h_padding, sizeof(int) * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(c_dims), h_dims, sizeof(int) * 10);

    // Create output tensor
    auto output = torch::zeros({N, C_out, D_out, H_out, W_out}, input.options());

    // Launch configuration
    int totalElements = N * C_out * D_out * H_out * W_out;
    int blockSize = 256;
    int gridSize = (totalElements + blockSize - 1) / blockSize;

    // Get raw pointers
    const float *input_ptr = input.data_ptr<float>();
    const float *weight_ptr = weight.data_ptr<float>();
    const float *bias_ptr = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float *output_ptr = output.data_ptr<float>();

    // Launch kernel
    conv_transposed_3d_cuda_kernel<<<gridSize, blockSize>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        totalElements,
        kD, kH, kW
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ConvTranspose3d forward with constant memory optimization",
          py::arg("input"),
          py::arg("weight"),
          py::arg("bias") = nullptr,
          py::arg("stride"),
          py::arg("padding"),
          py::arg("output_padding"),
          py::arg("groups"));
}