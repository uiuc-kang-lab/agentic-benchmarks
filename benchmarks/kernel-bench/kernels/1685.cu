#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void optimized_triangular_mm_kernel(const float* __restrict__ A,
                                             const float* __restrict__ B,
                                             float* __restrict__ C,
                                             int N) {
    const int warpSize = 32;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= N || col >= N) return;

    // Calculate based on triangular property
    if (row < col) {
        C[row * N + col] = 0.0f;
    } else {
        float sum = 0.0f;
        #pragma unroll 4
        // Exploit that only elements k from col to row contribute
        for (int k = col; k <= row; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Use larger blocks for greater occupancy if N is small
    const int threadsPerBlock = (N < 1024) ? 32 : 16;
    dim3 threads(threadsPerBlock, threadsPerBlock);
    dim3 blocks((N + threadsPerBlock - 1) / threadsPerBlock, (N + threadsPerBlock - 1) / threadsPerBlock);

    optimized_triangular_mm_kernel<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Triangular Matrix Multiplication (CUDA)");
}