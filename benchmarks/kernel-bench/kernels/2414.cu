#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel that uses shared memory to cache tiles of A and B for matrix multiplication
// C = A * B^T, where B is treated as transposed

const int TILE_SIZE = 32;

__global__ void shared_mem_matmul_transposed_kernel(const float* __restrict__ A,
                                                     const float* __restrict__ B,
                                                     float* __restrict__ C,
                                                     int M, int N, int K) {
    // Shared memory for tiles of A and B
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    // Calculate row and column index of the C element to work on
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    // Loop over tiles of A and B
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        // Load tiles into shared memory
        if (row < M && t * TILE_SIZE + threadIdx.x < K) {
            As[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_SIZE + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && t * TILE_SIZE + threadIdx.y < K) {
            Bs[threadIdx.y][threadIdx.x] = B[col * K + t * TILE_SIZE + threadIdx.y];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        // Compute partial product for this tile
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write the result to C
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

// Forward function callable from PyTorch
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.dim() == 2, "A must be 2D");
    TORCH_CHECK(B.dim() == 2, "B must be 2D");
    TORCH_CHECK(A.size(1) == B.size(1), "A and B must have the same K dimension");
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be on CUDA");
    TORCH_CHECK(A.is_contiguous() && B.is_contiguous(), "Inputs must be contiguous");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(0);

    auto C = torch::empty({M, N}, A.options());

    // Configure launch parameters:
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);

    // Launch the kernel
    shared_mem_matmul_transposed_kernel<<<grid, block>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, N, K
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Shared memory optimized matrix multiplication with transposed B (CUDA)");
}