#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// Allow block size tuning; default to 128 threads per block but can be overridden at compile time
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 128
#endif

// Optimized CUDA kernel for 2D transposed convolution with tunable block size
// Input tensor:  [batch, in_channels, in_height, in_width]
// Weight tensor: [in_channels, out_channels, kernel_size, kernel_size]
// Bias tensor:   [out_channels]
// Output tensor: [batch, out_channels, out_height, out_width]
// The kernel uses the __ldg() intrinsic for read-only global memory accesses
__global__ void conv_transpose2d_forward_kernel_blocksize(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch_size * out_channels * out_height * out_width;
  if (index >= total)
    return;

  // Decode flat index into (b, o, out_h, out_w)
  int w_out = index % out_width;
  int temp = index / out_width;
  int h_out = temp % out_height;
  temp /= out_height;
  int o = temp % out_channels;
  int b = temp / out_channels;

  // Use __ldg to load bias from read-only global memory
  float out_val = __ldg(&bias[o]);

  for (int c = 0; c < in_channels; ++c) {
    for (int p = 0; p < kernel_size; ++p) {
      int h_unscaled = h_out + padding - p * dilation;
      if (h_unscaled % stride != 0)
        continue;
      int h_in = h_unscaled / stride;
      if (h_in < 0 || h_in >= in_height)
        continue;
      for (int q = 0; q < kernel_size; ++q) {
        int w_unscaled = w_out + padding - q * dilation;
        if (w_unscaled % stride != 0)
          continue;
        int w_in = w_unscaled / stride;
        if (w_in < 0 || w_in >= in_width)
          continue;
        int input_idx = ((b * in_channels + c) * in_height + h_in) * in_width + w_in;
        int weight_idx = ((c * out_channels + o) * kernel_size + p) * kernel_size + q;
        out_val += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
      }
    }
  }

  int output_idx = ((b * out_channels + o) * out_height + h_out) * out_width + w_out;
  output[output_idx] = out_val;
}

// CUDA forward function with tunable block size
torch::Tensor conv_transpose2d_forward_cuda_opt_blocksize(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {
  
  // Input dimensions
  int batch_size = input.size(0);
  int in_channels = input.size(1);
  int in_height = input.size(2);
  int in_width = input.size(3);

  // Weight dimensions: [in_channels, out_channels, kernel_size, kernel_size]
  int out_channels = weight.size(1);
  int kernel_size = weight.size(2);  // assume square kernel

  // Compute output dimensions
  int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  int out_width  = (in_width - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  
  auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());

  int total_threads = batch_size * out_channels * out_height * out_width;
  int blocks = (total_threads + BLOCK_SIZE - 1) / BLOCK_SIZE;

  conv_transpose2d_forward_kernel_blocksize<<<blocks, BLOCK_SIZE>>>(
      input.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      output.data_ptr<float>(),
      batch_size,
      in_channels,
      out_channels,
      in_height,
      in_width,
      kernel_size,
      out_height,
      out_width,
      stride,
      padding,
      dilation);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in conv_transpose2d_forward_kernel_blocksize: %s\n", hipGetErrorString(err));
  }

  return output;
}

// Wrapper function handling possible None bias tensor
torch::Tensor conv_transpose2d_forward_wrapper_opt_blocksize(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,
    int stride,
    int padding,
    int dilation) {
  int out_channels = weight.size(1);
  torch::Tensor bias;
  if (bias_obj.is(pybind11::none())) {
    bias = torch::zeros({out_channels}, weight.options());
  } else {
    bias = bias_obj.cast<torch::Tensor>();
  }
  
  return conv_transpose2d_forward_cuda_opt_blocksize(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &conv_transpose2d_forward_wrapper_opt_blocksize,
        "ConvTranspose2d optimized forward (CUDA) with tunable block size",
        pybind11::arg("input"),
        pybind11::arg("weight"),
        pybind11::arg("bias"),
        pybind11::arg("stride"),
        pybind11::arg("padding"),
        pybind11::arg("dilation"));
}
