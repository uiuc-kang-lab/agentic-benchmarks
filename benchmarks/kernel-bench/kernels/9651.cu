#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

template <typename scalar_t>
__global__ void adaptiveDepthwiseConv2DKernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ w,
    const scalar_t* __restrict__ b,
    scalar_t* __restrict__ out,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int kernel_size,
    const int out_height,
    const int out_width,
    const int stride,
    const int padding,
    const bool use_tiled) {

    if (use_tiled) {
        const int w_out_idx = threadIdx.x + blockIdx.x * blockDim.x;
        const int h_out_idx = threadIdx.y + blockIdx.y * blockDim.y;
        const int c = blockIdx.z % in_channels;
        const int n = blockIdx.z / in_channels;

        if (w_out_idx >= out_width || h_out_idx >= out_height || n >= batch_size) {
            return;
        }

        scalar_t value = 0;
        
        #pragma unroll
        for (int kh = 0; kh < kernel_size; kh++) {
            int h_in = h_out_idx * stride - padding + kh;
            if (h_in >= 0 && h_in < in_height) {
                #pragma unroll
                for (int kw = 0; kw < kernel_size; kw++) {
                    int w_in = w_out_idx * stride - padding + kw;
                    if (w_in >= 0 && w_in < in_width) {
                        int x_index = ((n * in_channels + c) * in_height + h_in) * in_width + w_in;
                        int w_index = (c * kernel_size + kh) * kernel_size + kw;
                        value += __ldg(&x[x_index]) * w[w_index];
                    }
                }
            }
        }
        value += b[c];
        int out_index = ((n * in_channels + c) * out_height + h_out_idx) * out_width + w_out_idx;
        out[out_index] = value;

    } else {
        int total = batch_size * in_channels * out_height * out_width;
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int gridSize = blockDim.x * gridDim.x;

        for (; idx < total; idx += gridSize) {
            int w_out = idx % out_width;
            int tmp = idx / out_width;
            int h_out = tmp % out_height;
            tmp /= out_height;
            int c = tmp % in_channels;
            int n = tmp / in_channels;

            scalar_t value = 0;
            
            #pragma unroll
            for (int kh = 0; kh < kernel_size; kh++) {
                int h_in = h_out * stride - padding + kh;
                if (h_in >= 0 && h_in < in_height) {
                    #pragma unroll
                    for (int kw = 0; kw < kernel_size; kw++) {
                        int w_in = w_out * stride - padding + kw;
                        if (w_in >= 0 && w_in < in_width) {
                            int x_index = ((n * in_channels + c) * in_height + h_in) * in_width + w_in;
                            int w_index = (c * kernel_size + kh) * kernel_size + kw;
                            value += __ldg(&x[x_index]) * w[w_index];
                        }
                    }
                }
            }
            value += b[c];
            int out_index = ((n * in_channels + c) * out_height + h_out) * out_width + w_out;
            out[out_index] = value;
        }
    }
}

torch::Tensor forward_impl(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int groups) {

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);
    const int kernel_size = weight.size(2);
    const int out_height = (in_height + 2 * padding - kernel_size) / stride + 1;
    const int out_width  = (in_width + 2 * padding - kernel_size) / stride + 1;

    auto out = torch::empty({batch_size, in_channels, out_height, out_width}, x.options());

    const int total_elements = batch_size * in_channels * out_height * out_width;
    const bool use_tiled = total_elements <= 1048576;

    if (use_tiled) {
        dim3 threads(32, 16);
        dim3 blocks(
            (out_width + threads.x - 1) / threads.x,
            (out_height + threads.y - 1) / threads.y,
            batch_size * in_channels
        );
        AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "depthwise_conv2d_forward", ([&] {
            adaptiveDepthwiseConv2DKernel<scalar_t><<<blocks, threads>>>(
                x.data_ptr<scalar_t>(), weight.data_ptr<scalar_t>(),
                bias.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(),
                batch_size, in_channels, in_height, in_width,
                kernel_size, out_height, out_width,
                stride, padding, true
            );
        }));
    } else {
        const int threads = 256;
        const int blocks = (total_elements + threads - 1) / threads;
        AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "depthwise_conv2d_forward", ([&] {
            adaptiveDepthwiseConv2DKernel<scalar_t><<<blocks, threads>>>(
                x.data_ptr<scalar_t>(), weight.data_ptr<scalar_t>(),
                bias.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(),
                batch_size, in_channels, in_height, in_width,
                kernel_size, out_height, out_width,
                stride, padding, false
            );
        }));
    }

    return out;
}

namespace py = pybind11;

torch::Tensor forward_wrap(
    torch::Tensor x,
    torch::Tensor weight,
    py::object bias_obj,
    int stride,
    int padding,
    int groups) {
    torch::Tensor bias;
    if (bias_obj.is_none()) {
        bias = torch::zeros({x.size(1)}, x.options());
    } else {
        bias = bias_obj.cast<torch::Tensor>();
    }
    return forward_impl(x, weight, bias, stride, padding, groups);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        &forward_wrap,
        "Adaptive depthwise conv2d forward",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = 1,
        py::arg("padding") = 0,
        py::arg("groups") = 1
    );
}