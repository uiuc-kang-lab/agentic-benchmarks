#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Constant memory to store weights and bias
__constant__ float const_weight[1024]; // Ensure this size fits into the constant memory limit
__constant__ float const_bias[128];    // Maximum number of output channels

// CUDA kernel using constant memory for weights and bias for reducing global memory load time.
__global__ void conv2d_constant_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int in_height,
    int in_width,
    int out_channels,
    int kernel_height,
    int kernel_width,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation,
    int groups) {

    int w = blockIdx.x * blockDim.x + threadIdx.x;
    int h = blockIdx.y * blockDim.y + threadIdx.y;
    int oc = blockIdx.z;

    if (w < out_width && h < out_height && oc < out_channels) {
        for (int b = 0; b < batch_size; ++b) {
            float sum = 0.0f;
            int group_out_channels = out_channels / groups;
            int group = oc / group_out_channels;
            int in_channels_per_group = in_channels / groups;

            for (int c = 0; c < in_channels_per_group; ++c) {
                int input_channel = group * in_channels_per_group + c;
                for (int kh = 0; kh < kernel_height; ++kh) {
                    for (int kw = 0; kw < kernel_width; ++kw) {
                        int in_y = h * stride - padding + kh * dilation;
                        int in_x = w * stride - padding + kw * dilation;
                        if (in_y >= 0 && in_y < in_height && in_x >= 0 && in_x < in_width) {
                            int input_idx = ((b * in_channels + input_channel) * in_height + in_y) * in_width + in_x;
                            int weight_idx = ((oc * in_channels_per_group + c) * kernel_height + kh) * kernel_width + kw;
                            sum += input[input_idx] * const_weight[weight_idx];
                        }
                    }
                }
            }
            if (const_bias != nullptr) {
                sum += const_bias[oc];
            }
            int output_idx = ((b * out_channels + oc) * out_height + h) * out_width + w;
            output[output_idx] = sum;
        }
    }
}

// forward function initializes constant memory and launches the kernel
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int in_height = x.size(2);
    int in_width = x.size(3);
    int out_channels = weight.size(0);
    int kernel_height = weight.size(2);
    int kernel_width = weight.size(3);

    int out_height = (in_height + 2 * padding - dilation * (kernel_height - 1) - 1) / stride + 1;
    int out_width = (in_width + 2 * padding - dilation * (kernel_width - 1) - 1) / stride + 1;

    auto options = x.options();
    auto output = torch::empty({batch_size, out_channels, out_height, out_width}, options);

    const float* input_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();
    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        bias_ptr = bias.value().data_ptr<float>();
    }

    // Copy weights and bias to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(const_weight), weight_ptr, weight.numel() * sizeof(float));
    if (bias_ptr != nullptr) {
        hipMemcpyToSymbol(HIP_SYMBOL(const_bias), bias_ptr, bias->numel() * sizeof(float));
    }

    dim3 block_size(16, 16);
    dim3 grid_size((out_width + block_size.x - 1) / block_size.x,
                   (out_height + block_size.y - 1) / block_size.y,
                   out_channels);

    conv2d_constant_kernel<<<grid_size, block_size>>>(
        input_ptr,
        output_ptr,
        batch_size,
        in_channels,
        in_height,
        in_width,
        out_channels,
        kernel_height,
        kernel_width,
        out_height,
        out_width,
        stride,
        padding,
        dilation,
        groups
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA 2D Convolution with Constant Memory Usage");
}