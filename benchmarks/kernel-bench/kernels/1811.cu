#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32  // Increased tile size
#define NUM_STREAMS 4 // Increased number of streams
#define CHUNK_SIZE 2048 // Larger chunk size for better occupancy
#define WARP_SIZE 32

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   const int N,
                                   const int chunk_offset) {
    // Use vectorized loads where possible
    __shared__ float As[TILE_SIZE][TILE_SIZE+1]; // +1 for bank conflict avoidance
    __shared__ float Bs[TILE_SIZE][TILE_SIZE+1];
    
    const int row = blockIdx.y * blockDim.y + threadIdx.y + chunk_offset;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    float sum = 0.0f;
    
    // Early exit condition
    if (row >= N || col >= N) return;
    
    if (row < col) {
        C[row * N + col] = 0.0f;
        return;
    }

    // Calculate number of tiles needed
    const int num_tiles = (min(row, N-1) - col + TILE_SIZE) / TILE_SIZE;
    
    #pragma unroll 2
    for (int t = 0; t < num_tiles; t++) {
        const int tile_idx = col/TILE_SIZE + t;
        const int tile_offset = tile_idx * TILE_SIZE;

        if (row < N && (tile_offset + tx) <= row) {
            As[ty][tx] = A[row * N + tile_offset + tx];
        } else {
            As[ty][tx] = 0.0f;
        }
        
        if ((tile_offset + ty) < N && col < N) {
            Bs[ty][tx] = B[(tile_offset + ty) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        if (row >= col) {
            #pragma unroll 8
            for (int k = 0; k < TILE_SIZE; k++) {
                if ((tile_offset + k) >= col && (tile_offset + k) <= row) {
                    sum = fmaf(As[ty][k], Bs[k][tx], sum);
                }
            }
        }
        
        __syncthreads();
    }
    
    if (row >= col) {
        C[row * N + col] = sum;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D tensors");
    TORCH_CHECK(A.size(0) == A.size(1) && B.size(0) == B.size(1), "A and B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    const int N = A.size(0);
    auto C = torch::empty_like(A);

    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    
    hipHostRegister(A.data_ptr<float>(), N * N * sizeof(float), hipHostRegisterDefault);
    hipHostRegister(B.data_ptr<float>(), N * N * sizeof(float), hipHostRegisterDefault);
    
    for (int chunk = 0; chunk < N; chunk += CHUNK_SIZE) {
        const int chunk_rows = std::min(CHUNK_SIZE, N - chunk);
        dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE,
                      (chunk_rows + TILE_SIZE - 1) / TILE_SIZE);
        
        const int stream_idx = (chunk / CHUNK_SIZE) % NUM_STREAMS;
        
        triangular_mm_kernel<<<numBlocks, threadsPerBlock, 0, streams[stream_idx]>>>(
            A.data_ptr<float>(),
            B.data_ptr<float>(),
            C.data_ptr<float>(),
            N,
            chunk
        );
    }

    hipHostUnregister(A.data_ptr<float>());
    hipHostUnregister(B.data_ptr<float>());
    
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Hybrid chunked triangular matrix multiplication (CUDA)");
}