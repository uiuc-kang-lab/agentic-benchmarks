#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

__global__ void conv2d_cuda_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int N, int C_in, int H_in, int W_in,
    int C_out, int H_out, int W_out,
    int K_h, int K_w,
    int stride_h, int stride_w,
    int padding_h, int padding_w,
    int dilation_h, int dilation_w,
    int groups
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x; // Thread ID
    const int stride = blockDim.x * gridDim.x;
    const int total_elements = N * C_out * H_out * W_out;

    for (int index = tid; index < total_elements; index += stride) {
        const int w_out = index % W_out;
        int tmp = index / W_out;
        const int h_out = tmp % H_out;
        tmp = tmp / H_out;
        const int c_out = tmp % C_out;
        const int n = tmp / C_out;

        float value = (bias != nullptr) ? bias[c_out] : 0.0f;

        const int group = c_out / (C_out / groups);
        const int c_in_start = group * (C_in / groups);
        const int c_in_end = c_in_start + (C_in / groups);

        #pragma unroll 4
        for (int c_in = c_in_start; c_in < c_in_end; ++c_in) {
            for (int k_h = 0; k_h < K_h; ++k_h) {
                const int h_in = h_out * stride_h - padding_h + k_h * dilation_h;
                if (h_in >= 0 && h_in < H_in) {
                    for (int k_w = 0; k_w < K_w; ++k_w) {
                        const int w_in = w_out * stride_w - padding_w + k_w * dilation_w;
                        if (w_in >= 0 && w_in < W_in) {
                            const int input_idx = ((n * C_in + c_in) * H_in + h_in) * W_in + w_in;
                            const int weight_idx = (((c_out * (C_in / groups) + (c_in - c_in_start)) * K_h + k_h) * K_w) + k_w;
                            value += input[input_idx] * weight[weight_idx];
                        }
                    }
                }
            }
        }

        const int output_idx = ((n * C_out + c_out) * H_out + h_out) * W_out + w_out;
        output[output_idx] = value;
    }
}

torch::Tensor conv2d_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias_opt,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> dilation,
    int64_t groups
) {
    input = input.contiguous();
    weight = weight.contiguous();

    TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA");

    if (bias_opt.has_value()) {
        TORCH_CHECK(bias_opt.value().is_cuda(), "Bias tensor must be on CUDA if provided");
    }

    const int64_t N = input.size(0);
    const int64_t C_in = input.size(1);
    const int64_t H_in = input.size(2);
    const int64_t W_in = input.size(3);
    const int64_t C_out = weight.size(0);
    const int64_t K_h = weight.size(2);
    const int64_t K_w = weight.size(3);

    const int64_t stride_h = stride[0];
    const int64_t stride_w = stride[1];
    const int64_t padding_h = padding[0];
    const int64_t padding_w = padding[1];
    const int64_t dilation_h = dilation[0];
    const int64_t dilation_w = dilation[1];

    const int64_t H_out = (H_in + 2 * padding_h - dilation_h * (K_h - 1) - 1) / stride_h + 1;
    const int64_t W_out = (W_in + 2 * padding_w - dilation_w * (K_w - 1) - 1) / stride_w + 1;

    auto output = torch::zeros({N, C_out, H_out, W_out}, input.options());

    const float* input_ptr = input.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = nullptr;

    if (bias_opt.has_value()) {
        auto bias = bias_opt.value().contiguous();
        bias_ptr = bias.data_ptr<float>();
    }

    float* output_ptr = output.data_ptr<float>();

    const int threads_per_block = 256;
    const int num_blocks = std::min(65535, (int)((N * C_out * H_out * W_out + threads_per_block - 1) / threads_per_block));

    conv2d_cuda_kernel<<<num_blocks, threads_per_block>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        N, C_in, H_in, W_in,
        C_out, H_out, W_out,
        K_h, K_w,
        stride_h, stride_w,
        padding_h, padding_w,
        dilation_h, dilation_w,
        groups
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv2d_cuda, "Custom 2D convolution (CUDA)",
        py::arg("input"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = std::vector<int64_t>{1, 1},
        py::arg("padding") = std::vector<int64_t>{0, 0},
        py::arg("dilation") = std::vector<int64_t>{1, 1},
        py::arg("groups") = 1
    );
}