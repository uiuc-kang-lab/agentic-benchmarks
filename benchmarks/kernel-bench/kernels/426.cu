#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel for matrix-vector multiplication
template <typename scalar_t>
__global__ void matvec_mul_kernel(
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> A,
    const torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> B,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> C,
    int64_t M,
    int64_t K)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M)
    {
        // Compute partial sum in registers
        scalar_t local_sum = 0;
        
        // Coalesced memory access pattern for matrix A
        #pragma unroll 4
        for (int64_t k = 0; k < K; ++k)
        {
            local_sum += A[row][k] * B[k];
        }
        
        // Single write to global memory, no atomic needed
        C[row][0] = local_sum;
    }
}

// C++ function that wraps the CUDA kernel
torch::Tensor matvec_mul_cuda(torch::Tensor A, torch::Tensor B)
{
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");

    A = A.contiguous();
    B = B.contiguous();

    int64_t M = A.size(0);
    int64_t K = A.size(1);

    TORCH_CHECK(B.numel() == K, "B must have the same number of elements as columns in A");
    TORCH_CHECK(B.dim() == 1 || (B.dim() == 2 && B.size(1) == 1), "B must be a vector");

    auto B_flat = B.view({-1});
    auto C = torch::zeros({M, 1}, A.options());

    // Optimize thread block size for H100
    int threads = 512;
    int blocks = (M + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matvec_mul_cuda", ([&] {
        matvec_mul_kernel<scalar_t><<<blocks, threads>>>(
            A.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
            B_flat.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
            C.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
            M,
            K);
    }));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matvec_mul_cuda, "Matrix-Vector Multiplication (CUDA)");
}