#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Kernel definition unchanged
__global__ void max_pool2d_kernel(
    const float* input,
    float* output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    const int ow = output_idx % output_width;
    const int oh = (output_idx / output_width) % output_height;
    const int c = (output_idx / (output_width * output_height)) % channels;
    const int b = output_idx / (output_width * output_height * channels);

    float max_val = -std::numeric_limits<float>::infinity();

    for (int kh = 0; kh < kernel_size; kh++) {
        for (int kw = 0; kw < kernel_size; kw++) {
            const int ih = oh * stride - padding + kh * dilation;
            const int iw = ow * stride - padding + kw * dilation;

            if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
                const int input_idx = b * (channels * input_height * input_width) +
                                    c * (input_height * input_width) +
                                    ih * input_width +
                                    iw;
                max_val = max(max_val, input[input_idx]);
            }
        }
    }

    output[output_idx] = max_val;
}

// Forward function using CUDA streams
torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<<<blocks, threads, 0, stream1>>>(
            input.data_ptr<float>(),
            output.data_ptr<float>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
        hipMemcpyAsync(output.data_ptr<float>(), output.data_ptr<float>(), output.numel() * sizeof(float), hipMemcpyDeviceToDevice, stream2);
    }));

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}