#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

__constant__ int d_kernel_size;
__constant__ int d_stride;
__constant__ int d_padding;
__constant__ int d_dilation;

template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* input,
    scalar_t* output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width
) {
    const int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    const int ow = output_idx % output_width;
    const int oh = (output_idx / output_width) % output_height;
    const int c = (output_idx / (output_width * output_height)) % channels;
    const int b = output_idx / (output_width * output_height * channels);

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    for (int kh = 0; kh < d_kernel_size; kh++) {
        for (int kw = 0; kw < d_kernel_size; kw++) {
            const int ih = oh * d_stride - d_padding + kh * d_dilation;
            const int iw = ow * d_stride - d_padding + kw * d_dilation;

            if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
                const int input_idx = b * (channels * input_height * input_width) +
                                    c * (input_height * input_width) +
                                    ih * input_width +
                                    iw;
                max_val = max(max_val, input[input_idx]);
            }
        }
    }

    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    // Copy configuration to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_size), &kernel_size, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_stride), &stride, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_padding), &padding, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dilation), &dilation, sizeof(int));

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}