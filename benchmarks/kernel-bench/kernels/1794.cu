#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Tile size for shared memory optimization
#define TILE_SIZE 16
#define UNROLL_FACTOR 4

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = threadIdx.y;
    int tx = threadIdx.x;
    
    float sum = 0.0f;
    
    // Loop over tiles
    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Load tile into shared memory
        if (row < N && (t * TILE_SIZE + tx) <= row) {
            As[ty][tx] = A[row * N + (t * TILE_SIZE + tx)];
        } else {
            As[ty][tx] = 0.0f;
        }
        
        if (col < N && (t * TILE_SIZE + ty) >= col) {
            Bs[ty][tx] = B[(t * TILE_SIZE + ty) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        if (row < N && col < N && row >= col) {
            // Compute partial sum for this tile with loop unrolling
            #pragma unroll UNROLL_FACTOR
            for (int k = 0; k < TILE_SIZE; k++) {
                if ((t * TILE_SIZE + k) >= col && (t * TILE_SIZE + k) <= row) {
                    sum += As[ty][k] * Bs[k][tx];
                }
            }
        }
        
        __syncthreads();
    }
    
    if (row < N && col < N) {
        if (row < col) {
            C[row * N + col] = 0.0f;
        } else {
            C[row * N + col] = sum;
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE, 
                   (N + TILE_SIZE - 1) / TILE_SIZE);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}