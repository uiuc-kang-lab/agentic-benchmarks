#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

// Define block dimensions for optimized indexing
#define OFFSETS_PER_BLOCK 32
#define THREADS_FEATURE 8

// Optimized kernel with improved thread and block indexing

template <typename scalar_t>
__global__ void rms_norm_optimized_indexing_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int total_offsets,  // batch_size * numel_per_batch
    const int num_features,
    const int numel_per_batch,
    const float eps
) {
    // Calculate the global offset index corresponding to a (batch, offset) pair
    int global_offset = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_offset >= total_offsets) return;

    // Determine the batch id and the offset within the batch
    int batch_id = global_offset / numel_per_batch;
    int offset = global_offset % numel_per_batch;
    int base = batch_id * num_features * numel_per_batch;

    // Shared memory for reduction: size = OFFSETS_PER_BLOCK * THREADS_FEATURE
    __shared__ scalar_t sdata[OFFSETS_PER_BLOCK * THREADS_FEATURE];

    // Each thread in the column computes a partial sum over a subset of feature indices
    scalar_t partial_sum = 0;
    for (int f = threadIdx.y; f < num_features; f += THREADS_FEATURE) {
        int pos = base + f * numel_per_batch + offset;
        scalar_t val = input[pos];
        partial_sum += val * val;
    }

    // Store the partial sum in shared memory. Shared memory is laid out as [THREADS_FEATURE][OFFSETS_PER_BLOCK]
    int smem_index = threadIdx.y * OFFSETS_PER_BLOCK + threadIdx.x;
    sdata[smem_index] = partial_sum;
    __syncthreads();

    // Perform reduction along the feature dimension (vertical reduction within the column)
    for (int stride = THREADS_FEATURE / 2; stride > 0; stride /= 2) {
        if (threadIdx.y < stride) {
            sdata[smem_index] += sdata[(threadIdx.y + stride) * OFFSETS_PER_BLOCK + threadIdx.x];
        }
        __syncthreads();
    }

    // Thread with threadIdx.y == 0 in each column now holds the complete sum of squares
    scalar_t rms;
    if (threadIdx.y == 0) {
        scalar_t sumsq = sdata[threadIdx.x];
        rms = sqrt(sumsq / num_features + eps);
        // Store the computed rms in shared memory for use by all threads in this column
        sdata[threadIdx.x] = rms;
    }
    __syncthreads();
    rms = sdata[threadIdx.x];

    // Normalization: each thread in the column normalizes a subset of feature elements
    for (int f = threadIdx.y; f < num_features; f += THREADS_FEATURE) {
        int pos = base + f * numel_per_batch + offset;
        scalar_t val = input[pos];
        output[pos] = val / rms;
    }
}

// CUDA forward function with optimized indexing

torch::Tensor rms_norm_cuda_forward_optimized_indexing(torch::Tensor input, float eps) {
    auto output = torch::empty_like(input);

    const int batch_size = input.size(0);
    const int num_features = input.size(1);

    int numel_per_batch = 1;
    for (int i = 2; i < input.dim(); i++) {
        numel_per_batch *= input.size(i);
    }

    // Total number of (batch, offset) pairs to process
    int total_offsets = batch_size * numel_per_batch;

    // Define block dimensions: each block has OFFSETS_PER_BLOCK columns and THREADS_FEATURE rows
    dim3 block(OFFSETS_PER_BLOCK, THREADS_FEATURE);
    int grid_x = (total_offsets + OFFSETS_PER_BLOCK - 1) / OFFSETS_PER_BLOCK;
    dim3 grid(grid_x);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "rms_norm_cuda_optimized_indexing", ([&] {
        rms_norm_optimized_indexing_kernel<scalar_t><<<grid, block>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            total_offsets,
            num_features,
            numel_per_batch,
            eps
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &rms_norm_cuda_forward_optimized_indexing, "RMS normalization forward with optimized indexing (CUDA)");
}
