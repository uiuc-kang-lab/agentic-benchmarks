#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void conv2d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    float* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int out_channels,
    const int input_height,
    const int input_width,
    const int kernel_size,
    const int output_height,
    const int output_width,
    const int stride,
    const int padding) {
    
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int b = blockIdx.z;
    
    if (x >= output_width || y >= output_height || b >= batch_size) return;

    for (int oc = 0; oc < out_channels; ++oc) {
        float sum = 0.0f;

        for (int ic = 0; ic < in_channels; ++ic) {
            for (int kh = 0; kh < kernel_size; ++kh) {
                int ih = y * stride - padding + kh;
                if (ih >= 0 && ih < input_height) {
                    for (int kw = 0; kw < kernel_size; ++kw) {
                        int iw = x * stride - padding + kw;
                        if (iw >= 0 && iw < input_width) {
                            int input_idx = ((b * in_channels + ic) * input_height + ih) * input_width + iw;
                            int weight_idx = ((oc * in_channels + ic) * kernel_size + kh) * kernel_size + kw;
                            sum += input[input_idx] * weight[weight_idx];
                        }
                    }
                }
            }
        }

        int output_idx = ((b * out_channels + oc) * output_height + y) * output_width + x;
        output[output_idx] = sum;
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {
    
    TORCH_CHECK(x.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "Weight must be a CUDA tensor");
    if (bias.has_value()) {
        TORCH_CHECK(bias.value().is_cuda(), "Bias must be a CUDA tensor");
    }
    
    auto batch_size = x.size(0);
    auto in_channels = x.size(1);
    auto input_height = x.size(2);
    auto input_width = x.size(3);
    auto out_channels = weight.size(0);
    auto kernel_size = weight.size(2);
    
    auto output_height = (input_height + 2 * padding - kernel_size) / stride + 1;
    auto output_width = (input_width + 2 * padding - kernel_size) / stride + 1;
    
    auto output = torch::empty({batch_size, out_channels, output_height, output_width},
                             x.options());
    
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((output_width + threads.x - 1) / threads.x,
                (output_height + threads.y - 1) / threads.y,
                batch_size);
    
    conv2d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        out_channels,
        input_height,
        input_width,
        kernel_size,
        output_height,
        output_width,
        stride,
        padding);
    
    if (bias.has_value()) {
        output.add_(bias.value().view({1, -1, 1, 1}));
    }
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA optimized forward function for 2D convolution");
}
