#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

__global__ void einsum_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int BATCH, int I, int J, int L, int K
) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_idx >= BATCH * I * J * K) return;

    int k = global_idx % K;
    int remainder = global_idx / K;
    int j = remainder % J;
    remainder /= J;
    int i = remainder % I;
    int b = remainder / I;

    float sum = 0.0f;
    int a_base = b * I*J*L + i*J*L + j*L;
    int b_base = k;

    #pragma unroll 16
    for(int l = 0; l < L; ++l) {
        sum += A[a_base + l] * B[l*K + b_base];
    }
    
    C[global_idx] = sum;
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(A.dim() == 4, "A must be 4D");
    TORCH_CHECK(B.dim() == 2, "B must be 2D");
    TORCH_CHECK(A.size(3) == B.size(0), "Dimension mismatch in l");

    int BATCH = A.size(0), I = A.size(1), J = A.size(2), L = A.size(3);
    int K = B.size(1);
    
    auto C = torch::zeros({BATCH, I, J, K}, A.options());
    int total_elements = BATCH * I * J * K;
    
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;
    
    einsum_kernel<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        BATCH, I, J, L, K
    );
    
    return C;
}