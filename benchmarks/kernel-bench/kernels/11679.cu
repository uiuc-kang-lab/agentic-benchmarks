#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel for KL divergence with overlap of computation and memory transfers
__global__ void streamed_kl_div_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    // Simplified indexing for the kernel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    extern __shared__ float partial_sums[];
    float sum = 0.0f;
    
    for (int i = idx; i < n; i += stride) {
        float log_pred = log_predictions[i];
        float target = targets[i];
        sum += expf(log_pred) - target * log_pred;
    }
    
    partial_sums[threadIdx.x] = sum;
    __syncthreads();
    
    // Reduce within block
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            partial_sums[threadIdx.x] += partial_sums[threadIdx.x + offset];
        }
        __syncthreads();
    }
    
    if (threadIdx.x == 0) {
        atomicAdd(output, partial_sums[0]);
    }
}

// Host function leveraging CUDA streams
torch::Tensor streamed_kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());

    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;
    const int shared_mem = threads * sizeof(float);

    // Create CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocate device memory in the stream
    float *d_log_predictions, *d_targets, *d_output;
    hipMalloc((void**)&d_log_predictions, n * sizeof(float));
    hipMalloc((void**)&d_targets, n * sizeof(float));
    hipMalloc((void**)&d_output, sizeof(float));
    
    // Asynchronously copy data to device
    hipMemcpyAsync(d_log_predictions, log_predictions.data_ptr<float>(), n * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_targets, targets.data_ptr<float>(), n * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_output, output.data_ptr<float>(), sizeof(float), hipMemcpyHostToDevice, stream);
    
    // Launch kernel in stream
    streamed_kl_div_kernel<<<blocks, threads, shared_mem, stream>>>(
        d_log_predictions,
        d_targets,
        d_output,
        n
    );
    
    // Asynchronously copy the result back
    hipMemcpyAsync(output.data_ptr<float>(), d_output, sizeof(float), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    // Free memory
    hipFree(d_log_predictions);
    hipFree(d_targets);
    hipFree(d_output);
    hipStreamDestroy(stream);
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &streamed_kl_div_cuda_forward, "KL divergence with stream optimization (CUDA)");
}
