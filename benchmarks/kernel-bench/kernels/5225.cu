#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void coalesced_max_pool1d_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int64_t* __restrict__ indices,
    const int batch_size,
    const int num_channels,
    const int input_length,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation,
    const int output_length,
    bool return_indices)
{
    // Organize threads to ensure coalesced memory access
    const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    const int stride_x = blockDim.x * gridDim.x;
    
    // Process output elements in a way that ensures coalesced memory access
    for (int i = tidx; i < output_length; i += stride_x) {
        const int input_start = i * stride - padding;
        
        // Process all batches and channels for this output position
        for (int b = 0; b < batch_size; b++) {
            for (int c = 0; c < num_channels; c++) {
                float max_val = -INFINITY;
                int max_idx = -1;
                
                // Calculate aligned base index for coalesced memory access
                const int base_idx = (b * num_channels * input_length + c * input_length);
                
                // Process the pooling window
                #pragma unroll
                for (int k = 0; k < kernel_size; k++) {
                    const int pos = input_start + k * dilation;
                    if (pos >= 0 && pos < input_length) {
                        const float val = input[base_idx + pos];
                        if (val > max_val) {
                            max_val = val;
                            max_idx = pos;
                        }
                    }
                }
                
                // Write results with coalesced access pattern
                const int out_idx = (b * num_channels * output_length) + (c * output_length) + i;
                output[out_idx] = max_val;
                if (return_indices) {
                    indices[out_idx] = max_idx;
                }
            }
        }
    }
}

torch::Tensor forward(
    torch::Tensor x,
    int64_t kernel_size,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    bool return_indices)
{
    TORCH_CHECK(x.dim() == 3, "Input must be 3D");
    TORCH_CHECK(x.is_cuda(), "Input must be on CUDA");
    TORCH_CHECK(x.is_contiguous(), "Input must be contiguous");

    const int batch_size = x.size(0);
    const int num_channels = x.size(1);
    const int input_length = x.size(2);

    const int output_length = ((input_length + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    TORCH_CHECK(output_length > 0, "Output length must be positive");

    auto options = torch::TensorOptions().dtype(x.dtype()).device(x.device());
    auto output = torch::empty({batch_size, num_channels, output_length}, options);
    torch::Tensor indices;

    if (return_indices) {
        indices = torch::empty({batch_size, num_channels, output_length}, 
            options.dtype(torch::kInt64));
    }

    // Use 128 threads per block for good occupancy while maintaining coalescing
    const int threads_per_block = 128;
    // Calculate number of blocks needed to process output elements
    const int num_blocks = (output_length + threads_per_block - 1) / threads_per_block;
    
    coalesced_max_pool1d_kernel<<<num_blocks, threads_per_block>>>(
        x.data_ptr<float>(),
        output.data_ptr<float>(),
        return_indices ? indices.data_ptr<int64_t>() : nullptr,
        batch_size,
        num_channels,
        input_length,
        kernel_size,
        stride,
        padding,
        dilation,
        output_length,
        return_indices
    );

    return return_indices ? torch::cat({output, indices}, -1) : output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "MaxPool1D forward (CUDA)");
}