#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// CUDA kernel for 2D transposed convolution (conv_transpose2d) forward.
// Input tensor:  [batch, in_channels, in_height, in_width]
// Weight tensor: [in_channels, out_channels, kernel_size, kernel_size]
// Bias tensor:   [out_channels]
// Output tensor: [batch, out_channels, out_height, out_width]
//
// For each output element (b, o, out_h, out_w) we accumulate contributions:
//   For each input channel c and kernel element (p,q):
//     Let h_in = (out_h + padding - p * dilation) / stride
//         w_in = (out_w + padding - q * dilation) / stride
//     If the division is exact and the indices are in-bound, then add:
//         input[b, c, h_in, w_in] * weight[c, o, p, q]
//
// The bias for output channel o is added first.
__global__ void conv_transpose2d_forward_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch_size * out_channels * out_height * out_width;
  if (index >= total)
    return;

  // Decode index into (b, o, out_h, out_w)
  int w_out = index % out_width;
  int temp = index / out_width;
  int h_out = temp % out_height;
  temp /= out_height;
  int o = temp % out_channels;
  int b = temp / out_channels;

  float out_val = bias[o]; // start with bias

  for (int c = 0; c < in_channels; ++c) {
    for (int p = 0; p < kernel_size; ++p) {
      int h_unscaled = h_out + padding - p * dilation;
      if (h_unscaled % stride != 0)
        continue;
      int h_in = h_unscaled / stride;
      if (h_in < 0 || h_in >= in_height)
        continue;
      for (int q = 0; q < kernel_size; ++q) {
        int w_unscaled = w_out + padding - q * dilation;
        if (w_unscaled % stride != 0)
          continue;
        int w_in = w_unscaled / stride;
        if (w_in < 0 || w_in >= in_width)
          continue;
        int input_idx = ((b * in_channels + c) * in_height + h_in) * in_width + w_in;
        int weight_idx = ((c * out_channels + o) * kernel_size + p) * kernel_size + q;
        out_val += input[input_idx] * weight[weight_idx];
      }
    }
  }

  int output_idx = ((b * out_channels + o) * out_height + h_out) * out_width + w_out;
  output[output_idx] = out_val;
}

torch::Tensor conv_transpose2d_forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {
  
  // Get input dimensions.
  int batch_size = input.size(0);
  int in_channels = input.size(1);
  int in_height = input.size(2);
  int in_width = input.size(3);
  
  // Weight tensor: [in_channels, out_channels, kernel_size, kernel_size]
  int out_channels = weight.size(1);
  int kernel_size = weight.size(2);  // assume square kernel
  
  // Calculate output dimensions.
  int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  int out_width  = (in_width - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  
  auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());
  
  int total_threads = batch_size * out_channels * out_height * out_width;
  int threads = 1024;
  int blocks = (total_threads + threads - 1) / threads;
  
  conv_transpose2d_forward_kernel<<<blocks, threads>>>(
      input.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      output.data_ptr<float>(),
      batch_size,
      in_channels,
      out_channels,
      in_height,
      in_width,
      kernel_size,
      out_height,
      out_width,
      stride,
      padding,
      dilation);
  
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in conv_transpose2d_forward_kernel: %s\n", hipGetErrorString(err));
  }
  
  return output;
}

// Wrapper function to handle the possibility that the bias is None.
// If bias is None, we create a zero bias tensor of shape [out_channels].
torch::Tensor conv_transpose2d_forward_wrapper(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,  // using py::object to accept None
    int stride,
    int padding,
    int dilation) {
  
  int out_channels = weight.size(1);
  torch::Tensor bias;
  if (bias_obj.is(pybind11::none())) {
    bias = torch::zeros({out_channels}, weight.options());
  } else {
    bias = bias_obj.cast<torch::Tensor>();
  }
  
  return conv_transpose2d_forward_cuda(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &conv_transpose2d_forward_wrapper,
        "ConvTranspose2d forward (CUDA)",
        pybind11::arg("input"),
        pybind11::arg("weight"),
        pybind11::arg("bias"),
        pybind11::arg("stride"),
        pybind11::arg("padding"),
        pybind11::arg("dilation"));
}