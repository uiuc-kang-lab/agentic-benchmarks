#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

// This kernel uses shared memory to cache weights for each channel, reducing global memory access latency.
template <typename scalar_t>
__global__ void depthwiseConv2DKernelSharedMemory(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ w,
    const scalar_t* __restrict__ b,
    scalar_t* __restrict__ out,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int kernel_size,
    const int out_height,
    const int out_width,
    const int stride,
    const int padding) {

    // Shared memory for kernel weights
    extern __shared__ scalar_t shared_weights[];
    int bc = blockIdx.z;
    int c = bc % in_channels;
    int n = bc / in_channels;

    // Load weights into shared memory
    if (threadIdx.y == 0 && threadIdx.x < kernel_size * kernel_size) {
        shared_weights[threadIdx.x] = w[c * kernel_size * kernel_size + threadIdx.x];
    }
    __syncthreads();

    // Compute output spatial coordinates
    int h_out = blockIdx.y * blockDim.y + threadIdx.y;
    int w_out = blockIdx.x * blockDim.x + threadIdx.x;

    if (h_out < out_height && w_out < out_width) {
        const int batch_channel_offset = (n * in_channels + c);
        scalar_t value = 0;

        // Calculate input base position
        const int h_in_base = h_out * stride - padding;
        const int w_in_base = w_out * stride - padding;

        #pragma unroll
        for (int kh = 0; kh < kernel_size; ++kh) {
            const int h_in = h_in_base + kh;
            if (h_in >= 0 && h_in < in_height) {
                #pragma unroll
                for (int kw = 0; kw < kernel_size; ++kw) {
                    const int w_in = w_in_base + kw;
                    if (w_in >= 0 && w_in < in_width) {
                        const int x_index = (batch_channel_offset * in_height + h_in) * in_width + w_in;
                        const int w_index = kh * kernel_size + kw;
                        value += x[x_index] * shared_weights[w_index];
                    }
                }
            }
        }
        value += b[c];

        const int out_index = (batch_channel_offset * out_height + h_out) * out_width + w_out;
        out[out_index] = value;
    }
}

// Forward implementation using the shared memory kernel
torch::Tensor forward_impl(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int groups) {
    
    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);
    const int kernel_size = weight.size(2);
    const int out_height = (in_height + 2 * padding - kernel_size) / stride + 1;
    const int out_width  = (in_width  + 2 * padding - kernel_size) / stride + 1;

    auto out = torch::empty({batch_size, in_channels, out_height, out_width}, x.options());

    // Block dimensions: 32x8 threads per block
    const dim3 threads(32, 8);
    const dim3 blocks(
        (out_width + threads.x - 1) / threads.x,
        (out_height + threads.y - 1) / threads.y,
        batch_size * in_channels
    );

    // Shared memory size for kernel weights
    const int shared_memory_size = kernel_size * kernel_size * sizeof(scalar_t);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "depthwise_conv2d_forward_shared_memory", ([&] {
        depthwiseConv2DKernelSharedMemory<scalar_t><<<blocks, threads, shared_memory_size>>>(
            x.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            bias.data_ptr<scalar_t>(),
            out.data_ptr<scalar_t>(),
            batch_size, in_channels, in_height, in_width,
            kernel_size, out_height, out_width,
            stride, padding
        );
    }));

    return out;
}

// Wrap the forward implementation to handle optional bias.
torch::Tensor forward_wrap(
    torch::Tensor x,
    torch::Tensor weight,
    pybind11::object bias_obj,
    int stride,
    int padding,
    int groups) {
    torch::Tensor bias;
    if (bias_obj.is_none()) {
        bias = torch::zeros({x.size(1)}, x.options());
    } else {
        bias = bias_obj.cast<torch::Tensor>();
    }
    return forward_impl(x, weight, bias, stride, padding, groups);
}

namespace py = pybind11;

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        &forward_wrap,
        "Depthwise conv2d forward with shared memory optimization",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = 1,
        py::arg("padding") = 0,
        py::arg("groups") = 1
    );
}