#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Device function for main convolution computation
__global__ void compute_conv_transpose_ldg(
    const float* __restrict__ input, 
    const float* __restrict__ weight,
    float* __restrict__ output, 
    int in_channels, int out_channels,
    int kernel_size, int stride, int padding,
    int output_padding, int input_length) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int output_length = (input_length - 1) * stride - 2 * padding + 
                       kernel_size + output_padding;
    
    if (idx < output_length * out_channels) {
        int out_pos = idx / out_channels;
        int out_ch = idx % out_channels;
        float sum = 0.0f;

        for (int in_ch = 0; in_ch < in_channels; ++in_ch) {
            int weight_offset = (out_ch * in_channels + in_ch) * kernel_size;
            for (int k = 0; k < kernel_size; ++k) {
                int in_pos = (out_pos + padding - k) / stride;
                if (in_pos >= 0 && in_pos < input_length && 
                    (out_pos + padding - k) % stride == 0) {
                    sum += __ldg(&input[in_pos * in_channels + in_ch]) * 
                           __ldg(&weight[weight_offset + k]);
                }
            }
        }
        output[idx] = sum;
    }
}

torch::Tensor forward(
    const torch::Tensor& x,
    const torch::Tensor& weight,
    const c10::optional<torch::Tensor>& bias,
    int64_t stride,
    int64_t padding,
    int64_t output_padding,
    int64_t groups) {
    
    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
        auto result = torch::conv_transpose1d(
            x, weight, bias.value(),
            stride, padding, output_padding, groups
        );
        return result;
    }
    
    // Prepare for kernel launch
    const int input_length = x.size(2);
    const int in_channels = x.size(1);
    const int out_channels = weight.size(0);
    const int kernel_size = weight.size(2);
    const int output_length = (input_length - 1) * stride - 2 * padding + 
                              kernel_size + output_padding;
    
    auto output = torch::empty({x.size(0), out_channels, output_length}, x.options());
    
    int threads = 256;
    int blocks = (output_length * out_channels + threads - 1) / threads;
    
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    compute_conv_transpose_ldg<<<blocks, threads, 0, stream>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        output.data_ptr<float>(),
        in_channels, out_channels, kernel_size, stride, padding,
        output_padding, input_length
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Transposed 1D convolution forward (CUDA)");
}
