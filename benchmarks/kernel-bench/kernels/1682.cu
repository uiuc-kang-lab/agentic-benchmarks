#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized kernel using atomic operations only where necessary
__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                      const float* __restrict__ B,
                                      float* __restrict__ C,
                                      int N) {
    __shared__ float tileSum[32][32];
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = col; k <= row; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        tileSum[threadIdx.y][threadIdx.x] = sum;
        __syncthreads();

        // Only one thread writes the result to global memory
        if (threadIdx.x == 0) {
            float blockSum = 0.0f;
            for (int i = 0; i <= threadIdx.y; ++i) {
                blockSum += tileSum[i][threadIdx.y];
            }
            atomicAdd(&C[row * N + col], blockSum);
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::zeros_like(A);

    // Use 32x32 thread blocks to align with warp size
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((N + 31) / 32, (N + 31) / 32);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication with atomic optimizations (CUDA)");
}
