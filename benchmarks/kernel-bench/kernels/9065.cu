#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void conv1d_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int B,
    int in_channels,
    int in_size,
    int out_channels,
    int kernel_size,
    int out_size,
    int stride,
    int dilation
) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int total_threads = B * out_channels * out_size;
    
    if (tid >= total_threads) return;

    int o = tid % out_size;          // Output position (consecutive within warp)
    int oc = (tid / out_size) % out_channels;  // Output channel
    int b = tid / (out_size * out_channels);   // Batch

    int out_idx = b * (out_channels * out_size) + oc * out_size + o;
    float sum = 0.0f;

    int batch_offset = b * (in_channels * in_size);
    int weight_channel_offset = oc * (in_channels * kernel_size);

    for (int ic = 0; ic < in_channels; ++ic) {
        int input_channel_offset = batch_offset + ic * in_size;
        int weight_offset = weight_channel_offset + ic * kernel_size;

        for (int k = 0; k < kernel_size; ++k) {
            int input_pos = o * stride + k * dilation;
            if (input_pos < in_size) {
                sum += x[input_channel_offset + input_pos] * weight[weight_offset + k];
            }
        }
    }

    if (bias != nullptr) {
        sum += bias[oc];
    }

    output[out_idx] = sum;
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int dilation
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(weight.dim() == 3, "weight must be 3D");
    TORCH_CHECK(weight.size(1) == x.size(1), "Input channels mismatch");

    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        TORCH_CHECK(bias->dim() == 1, "bias must be 1D");
        TORCH_CHECK(bias->size(0) == weight.size(0), "Bias size mismatch");
    }

    int B = x.size(0);
    int in_channels = x.size(1);
    int in_size = x.size(2);
    int out_channels = weight.size(0);
    int kernel_size = weight.size(2);

    int out_size = (in_size - dilation * (kernel_size - 1) - 1) / stride + 1;
    TORCH_CHECK(out_size > 0, "Invalid output size");

    auto output = torch::empty({B, out_channels, out_size}, x.options());

    if (output.numel() == 0) return output;

    const float* x_data = x.data_ptr<float>();
    const float* weight_data = weight.data_ptr<float>();
    const float* bias_data = bias ? bias->data_ptr<float>() : nullptr;
    float* output_data = output.data_ptr<float>();

    int threads = 128;
    int total_elements = B * out_channels * out_size;
    int blocks = (total_elements + threads - 1) / threads;

    conv1d_kernel<<<blocks, threads>>>(
        x_data,
        weight_data,
        bias_data,
        output_data,
        B,
        in_channels,
        in_size,
        out_channels,
        kernel_size,
        out_size,
        stride,
        dilation
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel launch error: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "1D convolution forward (CUDA)");
}