#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

__global__ void multiplyKernelStreamed(const float* __restrict__ A,
                                        float* __restrict__ C,
                                        float s,
                                        int64_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        C[idx] = A[idx] * s;
    }
}

torch::Tensor forward(torch::Tensor A, float s)
{
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor.");
    TORCH_CHECK(A.scalar_type() == torch::kFloat, "Input tensor A must be of type float.");

    auto C = torch::empty_like(A);
    int64_t size = A.numel();
    const int threads = 256;
    const int blocks = (size + threads - 1) / threads;

    hipStream_t stream;
    hipStreamCreate(&stream);

    multiplyKernelStreamed<<<blocks, threads, 0, stream>>>(A.data_ptr<float>(),
                                                          C.data_ptr<float>(),
                                                          s,
                                                          size);

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix-scalar multiplication kernel with CUDA streams");
}