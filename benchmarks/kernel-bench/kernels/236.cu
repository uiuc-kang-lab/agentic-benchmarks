#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32  // Optimized tile size for shared memory/occupancy balance

__global__ void bmm_optimized_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int batch_size,
    int M,
    int K,
    int N
) {
    int b = blockIdx.z;
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    float sum = 0.0f;
    
    // Precompute batch pointers for faster address calculations
    const float* batch_A = A + b * M * K;
    const float* batch_B = B + b * K * N;
    
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Cooperative loading with combined bounds checking
        int loadA_col = t * TILE_SIZE + threadIdx.x;
        int loadB_row = t * TILE_SIZE + threadIdx.y;
        
        // As tile loading with thread.y frozen for better coalescing
        if (row < M && loadA_col < K) {
            As[threadIdx.y][threadIdx.x] = batch_A[row * K + loadA_col];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }

        // Bs tile loading with thread.x frozen for better coalescing
        if (loadB_row < K && col < N) {
            Bs[threadIdx.y][threadIdx.x] = batch_B[loadB_row * N + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();
        
        // Unrolled inner loop for reduced overhead
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        
        __syncthreads();
    }
    
    if (row < M && col < N) {
        C[b * M * N + row * N + col] = sum;
    }
}

torch::Tensor forward_bmm(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 3, "A must be 3D");
    TORCH_CHECK(B.dim() == 3, "B must be 3D");
    TORCH_CHECK(A.size(0) == B.size(0), "Batch sizes must match");
    TORCH_CHECK(A.size(2) == B.size(1), "Inner dimensions (K) must match");

    int batch_size = A.size(0);
    int M = A.size(1);
    int K = A.size(2);
    int N = B.size(2);

    auto C = torch::zeros({batch_size, M, N}, A.options());

    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid(
        (N + TILE_SIZE - 1) / TILE_SIZE,
        (M + TILE_SIZE - 1) / TILE_SIZE,
        batch_size
    );

    bmm_optimized_kernel<<<grid, block>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        batch_size, M, K, N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_bmm, "Optimized batched matrix multiplication (CUDA)");
}