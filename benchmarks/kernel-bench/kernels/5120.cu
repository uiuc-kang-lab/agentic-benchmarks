#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ATen/AccumulateType.h>

template <typename scalar_t>
__global__ void layernorm_forward_kernel_coalesced(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    const float eps,
    scalar_t* __restrict__ output,
    const int normalized_size) {

  const int instance_idx = blockIdx.x;
  const int tid = threadIdx.x;
  const int warp_size = 32;
  const int lane_id = tid % warp_size;
  const int warp_id = tid / warp_size;

  const scalar_t* in_ptr = input + instance_idx * normalized_size;
  scalar_t* out_ptr = output + instance_idx * normalized_size;

  using accscalar_t = at::acc_type<scalar_t, true>;

  extern __shared__ char smem[];
  accscalar_t* s_sum = reinterpret_cast<accscalar_t*>(smem);
  accscalar_t* s_sum_sq = s_sum + blockDim.x;

  // First pass: compute mean
  accscalar_t local_sum = 0;
  accscalar_t local_sum_sq = 0;

  // Ensure all elements are processed
  for (int i = tid; i < normalized_size; i += blockDim.x) {
    accscalar_t val = static_cast<accscalar_t>(in_ptr[i]);
    local_sum += val;
    local_sum_sq += val * val;
  }

  // Warp-level reduction
  #pragma unroll
  for (int offset = warp_size/2; offset > 0; offset /= 2) {
    local_sum += __shfl_down_sync(0xffffffff, local_sum, offset);
    local_sum_sq += __shfl_down_sync(0xffffffff, local_sum_sq, offset);
  }

  // Store warp results to shared memory
  if (lane_id == 0) {
    s_sum[warp_id] = local_sum;
    s_sum_sq[warp_id] = local_sum_sq;
  }
  __syncthreads();

  // Final reduction across warps
  if (tid < warp_size) {
    const int num_warps = blockDim.x / warp_size;
    if (tid < num_warps) {
      local_sum = s_sum[tid];
      local_sum_sq = s_sum_sq[tid];
    } else {
      local_sum = 0;
      local_sum_sq = 0;
    }

    #pragma unroll
    for (int offset = warp_size/2; offset > 0; offset /= 2) {
      local_sum += __shfl_down_sync(0xffffffff, local_sum, offset);
      local_sum_sq += __shfl_down_sync(0xffffffff, local_sum_sq, offset);
    }
  }

  __shared__ accscalar_t mean, inv_std;
  if (tid == 0) {
    mean = local_sum / normalized_size;
    // Compute variance in a numerically stable way
    accscalar_t var = fmax(local_sum_sq / normalized_size - mean * mean, 0.0f);
    inv_std = rsqrt(var + static_cast<accscalar_t>(eps));
  }
  __syncthreads();

  // Second pass: normalize with coalesced access
  for (int i = tid; i < normalized_size; i += blockDim.x) {
    accscalar_t val = static_cast<accscalar_t>(in_ptr[i]);
    accscalar_t norm_val = (val - mean) * inv_std;
    out_ptr[i] = static_cast<scalar_t>(
        norm_val * static_cast<accscalar_t>(weight[i]) +
        static_cast<accscalar_t>(bias[i]));
  }
}

torch::Tensor layernorm_forward(torch::Tensor x, torch::Tensor weight, torch::Tensor bias, double eps = 1e-5) {
  auto output = torch::empty_like(x);
  int normalized_size = weight.numel();
  int outer_size = x.numel() / normalized_size;

  const int warp_size = 32;
  const int warps_per_block = 8;
  const int threads = warp_size * warps_per_block;
  int blocks = outer_size;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "layernorm_forward_cuda", ([&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    int shared_size = threads * 2 * sizeof(accscalar_t);
    layernorm_forward_kernel_coalesced<scalar_t><<<blocks, threads, shared_size>>>(
        x.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(),
        bias.data_ptr<scalar_t>(),
        static_cast<float>(eps),
        output.data_ptr<scalar_t>(),
        normalized_size);
  }));

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &layernorm_forward, "LayerNorm forward (CUDA)",
        py::arg("x"), py::arg("weight"), py::arg("bias"), py::arg("eps") = 1e-5);
}