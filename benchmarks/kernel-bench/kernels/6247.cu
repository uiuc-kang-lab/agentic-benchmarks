#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template<int BLOCK_W = 32, int BLOCK_H = 8>
__global__ void avg_pool3d_forward_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int batch_size, int channels,
    int in_d, int in_h, int in_w,
    int out_d, int out_h, int out_w,
    int kernel_size, int stride, int padding) {

    // Shared memory to cache input tiles
    extern __shared__ float shared_input[];
    
    // Use blockIdx.z for (n, c, d_out) combined
    int idx = blockIdx.z;
    int d_out = idx % out_d;
    idx /= out_d;
    int c = idx % channels;
    int n = idx / channels;

    int h_out = blockIdx.y * BLOCK_H + threadIdx.y;
    int w_out = blockIdx.x * BLOCK_W + threadIdx.x;

    // Early exit if outside output bounds
    if (h_out >= out_h || w_out >= out_w) return;

    // Calculate input region needed for this thread block
    int h_in_start = blockIdx.y * BLOCK_H * stride - padding;
    int w_in_start = blockIdx.x * BLOCK_W * stride - padding;
    
    // Calculate shared memory dimensions including padding for pooling window
    int shared_h = BLOCK_H * stride + kernel_size;
    int shared_w = BLOCK_W * stride + kernel_size;

    // Calculate input boundaries once
    int d_start = d_out * stride - padding;
    int d_end = min(d_start + kernel_size, in_d);
    d_start = max(0, d_start);

    int h_load_start = max(0, h_in_start);
    int h_load_end = min(h_in_start + shared_h, in_h);
    int w_load_start = max(0, w_in_start);
    int w_load_end = min(w_in_start + shared_w, in_w);

    // Initialize shared memory to zero
    for (int h = threadIdx.y; h < shared_h; h += BLOCK_H) {
        for (int w = threadIdx.x; w < shared_w; w += BLOCK_W) {
            for (int d = 0; d < kernel_size; d++) {
                shared_input[((d) * shared_h + h) * shared_w + w] = 0.0f;
            }
        }
    }
    __syncthreads();

    // Load input data into shared memory - only valid regions
    for (int d = d_start; d < d_end; d++) {
        for (int h = h_load_start; h < h_load_end; h += BLOCK_H) {
            int h_shared = h - h_in_start;
            if (threadIdx.y + h < h_load_end) {
                for (int w = w_load_start; w < w_load_end; w += BLOCK_W) {
                    if (threadIdx.x + w < w_load_end) {
                        int input_idx = (((n * channels + c) * in_d + d) * in_h + h) * in_w + w;
                        int shared_idx = ((d - d_start) * shared_h + h_shared) * shared_w + (w - w_in_start);
                        shared_input[shared_idx] = input[input_idx];
                    }
                }
            }
        }
    }

    __syncthreads();

    // Compute pooling window boundaries for this output element
    int h_start = h_out * stride - padding;
    int w_start = w_out * stride - padding;
    
    float sum = 0.0f;
    int count = 0;
    
    // Compute pooling using shared memory with reduced divergence
    int h_pool_start = max(0, h_start);
    int h_pool_end = min(h_start + kernel_size, in_h);
    int w_pool_start = max(0, w_start);
    int w_pool_end = min(w_start + kernel_size, in_w);

    // Compute pooling using shared memory - boundaries already handled
    for (int d = d_start; d < d_end; d++) {
        for (int h = h_pool_start; h < h_pool_end; h++) {
            int h_shared = h - h_in_start;
            for (int w = w_pool_start; w < w_pool_end; w++) {
                int w_shared = w - w_in_start;
                int shared_idx = ((d - d_start) * shared_h + h_shared) * shared_w + w_shared;
                sum += shared_input[shared_idx];
                count++;
            }
        }
    }

    // Write output
    int pool_volume = kernel_size * kernel_size * kernel_size;
    int output_idx = ((((n * channels + c) * out_d + d_out) * out_h + h_out) * out_w + w_out);
    output[output_idx] = sum / static_cast<float>(pool_volume);
}

at::Tensor forward(at::Tensor input, int kernel_size, int stride, int padding) {
    TORCH_CHECK(input.dim() == 5, "Input tensor must be 5-dimensional");
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    int batch_size = input.size(0);
    int channels = input.size(1);
    int in_d = input.size(2);
    int in_h = input.size(3);
    int in_w = input.size(4);

    int out_d = (in_d + 2 * padding - kernel_size) / stride + 1;
    int out_h = (in_h + 2 * padding - kernel_size) / stride + 1;
    int out_w = (in_w + 2 * padding - kernel_size) / stride + 1;

    auto output = at::empty({batch_size, channels, out_d, out_h, out_w}, input.options());

    constexpr int BLOCK_W = 32;
    constexpr int BLOCK_H = 8;
    
    dim3 block(BLOCK_W, BLOCK_H);
    dim3 grid((out_w + BLOCK_W - 1) / BLOCK_W,
              (out_h + BLOCK_H - 1) / BLOCK_H,
              batch_size * channels * out_d);

    // Calculate shared memory size
    int shared_h = BLOCK_H * stride + kernel_size;
    int shared_w = BLOCK_W * stride + kernel_size;
    int shared_mem_size = kernel_size * shared_h * shared_w * sizeof(float);

    avg_pool3d_forward_kernel<BLOCK_W, BLOCK_H><<<grid, block, shared_mem_size>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size, channels,
        in_d, in_h, in_w,
        out_d, out_h, out_w,
        kernel_size, stride, padding);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed : ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D Average Pooling forward (CUDA) with shared memory");
}