#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>
#include <c10/cuda/CUDAStream.h>

// Constant memory for read-only data
__constant__ int const_outer;
__constant__ int const_inner;
__constant__ int const_r;

// Each warp computes the min reduction for one output element using warp-level primitives
// The input is logically reshaped as [outer, r, inner], and reduction is performed along the r dimension.
template <typename scalar_t>
__global__ void min_reduce_warp_const_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output) {
  // Compute global warp id: each warp is responsible for one output element
  int warpId = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  if (warpId >= const_outer * const_inner) return;

  int outer_idx = warpId / const_inner;
  int inner_idx = warpId % const_inner;
  int base = outer_idx * (const_r * const_inner) + inner_idx;

  // Compute lane id within the warp
  int lane = threadIdx.x % 32;

  // Each thread computes a partial min over the reduction dimension with stride = warpSize
  scalar_t my_min = std::numeric_limits<scalar_t>::max();
  for (int j = lane; j < const_r; j += 32) {
    int pos = base + j * const_inner;
    scalar_t val = input[pos];
    if (val < my_min) {
      my_min = val;
    }
  }

  // Warp-level reduction using __shfl_down_sync to combine the results
  for (int offset = 16; offset > 0; offset /= 2) {
    scalar_t other = __shfl_down_sync(0xffffffff, my_min, offset);
    if (other < my_min) {
      my_min = other;
    }
  }

  // The first lane of the warp writes the result
  if (lane == 0) {
    output[warpId] = my_min;
  }
}

// Host function to set up the kernel launch
torch::Tensor forward(torch::Tensor input, int64_t dim) {
  TORCH_CHECK(input.is_cuda(), "input must be a CUDA tensor");
  if (!input.is_contiguous()) {
    input = input.contiguous();
  }

  int ndim = input.dim();
  TORCH_CHECK(dim >= 0 && dim < ndim, "dim out of range");

  // Calculate sizes: outer dimensions, size of reduction dimension (r), and inner dimensions
  int outer = 1;
  for (int i = 0; i < dim; i++) {
    outer *= input.size(i);
  }
  int r = input.size(dim);
  int inner = 1;
  for (int i = dim + 1; i < ndim; i++) {
    inner *= input.size(i);
  }

  // Create the output shape by removing the reduced dimension
  std::vector<int64_t> output_shape;
  for (int i = 0; i < ndim; i++) {
    if (i != dim) {
      output_shape.push_back(input.size(i));
    }
  }
  auto output = torch::empty(output_shape, input.options());

  // Copy constants to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(const_outer), &outer, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(const_inner), &inner, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(const_r), &r, sizeof(int));

  // Each output element is processed by one warp (32 threads)
  int total_warps = outer * inner;
  int threads_per_block = 128;  // 128 threads per block gives 4 warps per block
  int num_blocks = (total_warps * 32 + threads_per_block - 1) / threads_per_block;

  AT_DISPATCH_ALL_TYPES(input.scalar_type(), "min_reduce_warp_const_cuda", ([&] {
    min_reduce_warp_const_kernel<scalar_t><<<num_blocks, threads_per_block, 0,
      c10::cuda::getCurrentCUDAStream().stream()>>>(
        input.data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>());
  }));

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "Min reduction over a specified dimension using constant memory (CUDA)");
}