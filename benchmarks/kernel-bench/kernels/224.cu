#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Define tile size for shared memory tiling
#define TILE_SIZE 16

// CUDA kernel for batched matrix multiplication using shared memory tiling
// Computes C = A * B for each batch.
// A: (batch_size, M, K), B: (batch_size, K, N), C: (batch_size, M, N)
__global__ void bmm_tiled_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int batch_size,
    int M,
    int K,
    int N
) {
    // Batch index from grid z dimension
    int b = blockIdx.z;

    // Compute row and column indices in the C matrix
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    float value = 0.0f;
    
    // Declare shared memory for tiles of A and B
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    // Loop over tiles of the input matrices
    int numTiles = (K + TILE_SIZE - 1) / TILE_SIZE;
    for (int t = 0; t < numTiles; t++) {
        // Load a tile of A into shared memory
        int aCol = t * TILE_SIZE + threadIdx.x;
        if (row < M && aCol < K) {
            As[threadIdx.y][threadIdx.x] = A[b * M * K + row * K + aCol];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }

        // Load a tile of B into shared memory
        int bRow = t * TILE_SIZE + threadIdx.y;
        if (col < N && bRow < K) {
            Bs[threadIdx.y][threadIdx.x] = B[b * K * N + bRow * N + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        // Multiply the two tiles together
        for (int k = 0; k < TILE_SIZE; k++) {
            value += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write the result in C if within bounds
    if (row < M && col < N) {
        C[b * M * N + row * N + col] = value;
    }
}

// Forward function to launch the kernel
torch::Tensor forward_bmm(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 3, "A must be 3D");
    TORCH_CHECK(B.dim() == 3, "B must be 3D");
    TORCH_CHECK(A.size(0) == B.size(0), "Batch sizes must match");
    TORCH_CHECK(A.size(2) == B.size(1), "Inner dimensions (K) must match");

    int batch_size = A.size(0);
    int M = A.size(1);
    int K = A.size(2);
    int N = B.size(2);

    auto options = torch::TensorOptions().dtype(A.dtype()).device(A.device());
    auto C = torch::zeros({batch_size, M, N}, options);

    // Configure block and grid dimensions
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE, batch_size);

    bmm_tiled_kernel<<<grid, block>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        batch_size, M, K, N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_bmm, "Batched matrix multiplication with tiling (CUDA)");
}
