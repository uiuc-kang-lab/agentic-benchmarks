#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ATen/AccumulateType.h>

static const int NUM_STREAMS = 4;
static hipStream_t streams[NUM_STREAMS];
static bool streams_created = false;

// Constant memory for weight and bias (64KB limit)
__constant__ float c_weight[32768];  // 32K floats
__constant__ float c_bias[32768];    // 32K floats

void create_streams() {
    if (!streams_created) {
        for (int i = 0; i < NUM_STREAMS; i++) {
            hipStreamCreate(&streams[i]);
        }
        streams_created = true;
    }
}

void destroy_streams() {
    if (streams_created) {
        for (int i = 0; i < NUM_STREAMS; i++) {
            hipStreamDestroy(&streams[i]);
        }
        streams_created = false;
    }
}

template <typename scalar_t>
__global__ void layernorm_constant_kernel(
    const scalar_t* __restrict__ input,
    const float eps,
    scalar_t* __restrict__ output,
    const int normalized_size,
    const int chunk_size,
    const int chunk_offset) {

    using accscalar_t = at::acc_type<scalar_t, true>;
    
    const int tidx = threadIdx.x;
    const int tidy = threadIdx.y;
    const int instance_idx = blockIdx.x + chunk_offset;
    
    extern __shared__ char smem[];
    accscalar_t* s_sum = reinterpret_cast<accscalar_t*>(smem);
    accscalar_t* s_sum_sq = s_sum + blockDim.x * blockDim.y;
    
    const scalar_t* in_ptr = input + instance_idx * normalized_size;
    scalar_t* out_ptr = output + instance_idx * normalized_size;
    
    const int thread_stride = blockDim.x * blockDim.y;
    const int thread_id = tidy * blockDim.x + tidx;
    
    // Use vectorized loads for better memory throughput
    const int vector_size = 4;
    const int vector_limit = normalized_size & ~(vector_size - 1);
    
    accscalar_t local_sum = 0;
    accscalar_t local_sum_sq = 0;
    
    #pragma unroll 4
    for (int idx = thread_id * vector_size; idx < vector_limit; idx += thread_stride * vector_size) {
        accscalar_t vals[vector_size];
        #pragma unroll
        for (int j = 0; j < vector_size; j++) {
            vals[j] = static_cast<accscalar_t>(in_ptr[idx + j]);
            local_sum += vals[j];
            local_sum_sq += vals[j] * vals[j];
        }
    }
    
    // Handle remaining elements
    for (int idx = vector_limit + thread_id; idx < normalized_size; idx += thread_stride) {
        accscalar_t val = static_cast<accscalar_t>(in_ptr[idx]);
        local_sum += val;
        local_sum_sq += val * val;
    }
    
    s_sum[thread_id] = local_sum;
    s_sum_sq[thread_id] = local_sum_sq;
    __syncthreads();
    
    if (thread_id < 32) {
        accscalar_t warp_sum = 0;
        accscalar_t warp_sum_sq = 0;
        
        #pragma unroll
        for (int i = thread_id; i < thread_stride; i += 32) {
            warp_sum += s_sum[i];
            warp_sum_sq += s_sum_sq[i];
        }
        
        #pragma unroll
        for (int offset = 16; offset > 0; offset /= 2) {
            warp_sum += __shfl_down_sync(0xffffffff, warp_sum, offset);
            warp_sum_sq += __shfl_down_sync(0xffffffff, warp_sum_sq, offset);
        }
        
        if (thread_id == 0) {
            s_sum[0] = warp_sum;
            s_sum_sq[0] = warp_sum_sq;
        }
    }
    __syncthreads();
    
    __shared__ accscalar_t mean, inv_std;
    if (thread_id == 0) {
        mean = s_sum[0] / normalized_size;
        accscalar_t variance = (s_sum_sq[0] / normalized_size) - (mean * mean);
        inv_std = rsqrt(variance + static_cast<accscalar_t>(eps));
    }
    __syncthreads();
    
    #pragma unroll 4
    for (int idx = thread_id * vector_size; idx < vector_limit; idx += thread_stride * vector_size) {
        accscalar_t vals[vector_size];
        #pragma unroll
        for (int j = 0; j < vector_size; j++) {
            vals[j] = static_cast<accscalar_t>(in_ptr[idx + j]);
            vals[j] = (vals[j] - mean) * inv_std;
            out_ptr[idx + j] = static_cast<scalar_t>(
                vals[j] * static_cast<accscalar_t>(c_weight[idx + j]) + 
                static_cast<accscalar_t>(c_bias[idx + j]));
        }
    }
    
    for (int idx = vector_limit + thread_id; idx < normalized_size; idx += thread_stride) {
        accscalar_t val = static_cast<accscalar_t>(in_ptr[idx]);
        val = (val - mean) * inv_std;
        out_ptr[idx] = static_cast<scalar_t>(
            val * static_cast<accscalar_t>(c_weight[idx]) + 
            static_cast<accscalar_t>(c_bias[idx]));
    }
}

torch::Tensor layernorm_forward(torch::Tensor x, torch::Tensor weight, torch::Tensor bias, double eps = 1e-5) {
    create_streams();
    
    auto output = torch::empty_like(x);
    
    const int normalized_size = weight.numel();
    const int outer_size = x.numel() / normalized_size;
    const int chunk_size = (outer_size + NUM_STREAMS - 1) / NUM_STREAMS;
    
    // Copy weight and bias to constant memory
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_weight), weight.data_ptr<float>(), 
                                 normalized_size * sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_bias), bias.data_ptr<float>(), 
                                 normalized_size * sizeof(float)));
    
    const dim3 threads(32, 32);
    const int shared_mem_size = threads.x * threads.y * 2 * sizeof(float);
    
    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "layernorm_forward_cuda", ([&] {
        for (int i = 0; i < NUM_STREAMS; i++) {
            int stream_chunk_size = std::min(chunk_size, outer_size - i * chunk_size);
            if (stream_chunk_size <= 0) break;
            
            const dim3 blocks(stream_chunk_size);
            
            layernorm_constant_kernel<scalar_t><<<blocks, threads, shared_mem_size, streams[i]>>>(
                x.data_ptr<scalar_t>(),
                static_cast<float>(eps),
                output.data_ptr<scalar_t>(),
                normalized_size,
                chunk_size,
                i * chunk_size);
        }
    }));
    
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &layernorm_forward, "LayerNorm forward (CUDA)",
          py::arg("x"), py::arg("weight"), py::arg("bias"), py::arg("eps") = 1e-5);
    m.def("cleanup", &destroy_streams, "Cleanup CUDA streams");
}