#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Kernel leveraging shared memory with prefetching
__global__ void leaky_relu_kernel_prefetch(const float* x, float* out, float negative_slope, int n) {
    extern __shared__ float shared_x[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Load data into shared memory, prefetching to reduce latency
    if (idx < n) {
        shared_x[tid] = x[idx];
    }
    __syncthreads();

    // Apply LeakyReLU with data in shared memory
    if (idx < n) {
        float val = shared_x[tid];
        out[idx] = val > 0.0f ? val : val * negative_slope;
    }
}

torch::Tensor leaky_relu_forward_prefetch(torch::Tensor x, float negative_slope) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int n = x.numel();

    const int threads = 512;  // Adjust threads for optimal performance
    const int blocks = (n + threads - 1) / threads;
    size_t shared_memory_size = threads * sizeof(float);

    leaky_relu_kernel_prefetch<<<blocks, threads, shared_memory_size>>>(
        x.data_ptr<float>(), out.data_ptr<float>(), negative_slope, n
    );

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &leaky_relu_forward_prefetch, "LeakyReLU forward with shared memory prefetching (CUDA)");
}