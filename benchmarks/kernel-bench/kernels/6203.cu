#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized device function to compute the sum over a pooling window in the input
template<int kernel_size>
__device__ inline float compute_window_sum_optimized(const float* __restrict__ input,
                                                     int n, int c,
                                                     int d_out, int h_out, int w_out,
                                                     int stride, int padding,
                                                     int in_d, int in_h, int in_w,
                                                     int channels) {
    int d_start = d_out * stride - padding;
    int h_start = h_out * stride - padding;
    int w_start = w_out * stride - padding;

    int d_end = d_start + kernel_size;
    int h_end = h_start + kernel_size;
    int w_end = w_start + kernel_size;

    int d_start_clamped = (d_start < 0) ? 0 : d_start;
    int h_start_clamped = (h_start < 0) ? 0 : h_start;
    int w_start_clamped = (w_start < 0) ? 0 : w_start;
    int d_end_clamped = (d_end > in_d) ? in_d : d_end;
    int h_end_clamped = (h_end > in_h) ? in_h : h_end;
    int w_end_clamped = (w_end > in_w) ? in_w : w_end;

    float sum = 0.0f;
    for (int d = d_start_clamped; d < d_end_clamped; ++d) {
        for (int h = h_start_clamped; h < h_end_clamped; ++h) {
            int base_index = (((n * channels + c) * in_d + d) * in_h + h) * in_w;
            for (int w = w_start_clamped; w < w_end_clamped; ++w) {
                // Use __ldg to read from read-only cache and align loads
                sum += __ldg(&input[base_index + w]);
            }
        }
    }
    return sum;
}

// Average pooling kernel with memory optimizations
__global__ void avg_pool3d_memory_optimized_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int batch_size,
    int channels,
    int in_d, int in_h, int in_w,
    int out_d, int out_h, int out_w,
    int kernel_size, int stride, int padding) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = batch_size * channels * out_d * out_h * out_w;

    while (index < total_elements) {
        int n, c, d_out, h_out, w_out;
        decompose_index(index, out_w, out_h, out_d, channels, n, c, d_out, h_out, w_out);

        float sum = compute_window_sum_optimized<3>(input, n, c, d_out, h_out, w_out, // Assuming kernel_size is 3
                                         stride, padding,
                                         in_d, in_h, in_w, channels);
        int pool_volume = kernel_size * kernel_size * kernel_size;
        output[index] = sum / static_cast<float>(pool_volume);

        index += blockDim.x * gridDim.x;
    }
}

at::Tensor forward(at::Tensor input, int kernel_size, int stride, int padding) {
    // Ensure input is a 5D CUDA tensor
    TORCH_CHECK(input.dim() == 5, "Input tensor must be 5-dimensional");
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    int batch_size = input.size(0);
    int channels   = input.size(1);
    int in_d       = input.size(2);
    int in_h       = input.size(3);
    int in_w       = input.size(4);

    // Compute the output dimensions
    int out_d = (in_d + 2 * padding - kernel_size) / stride + 1;
    int out_h = (in_h + 2 * padding - kernel_size) / stride + 1;
    int out_w = (in_w + 2 * padding - kernel_size) / stride + 1;

    auto output = at::empty({batch_size, channels, out_d, out_h, out_w}, input.options());

    int total_elements = batch_size * channels * out_d * out_h * out_w;
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;

    const float* input_ptr = input.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();

    avg_pool3d_memory_optimized_kernel<<<blocks, threads>>>(input_ptr, output_ptr,
                                                    batch_size, channels,
                                                    in_d, in_h, in_w,
                                                    out_d, out_h, out_w,
                                                    kernel_size, stride, padding);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed : ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D Average Pooling forward (CUDA) - memory optimized version");
}
