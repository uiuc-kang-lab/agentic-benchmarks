#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device function for computing average pooling for a single output element
__device__ float compute_avg_pool1d(
    const float* __restrict__ input,
    const int kernel_size,
    const int stride,
    const int padding,
    const int input_length,
    const int start_idx,
    const int input_base) {

    float sum = 0.0f;

    #pragma unroll
    for (int k = 0; k < kernel_size; ++k) {
        const int pos_input = start_idx + k;
        if (pos_input >= 0 && pos_input < input_length) {
            sum += input[input_base + pos_input];
        }
    }

    return sum / kernel_size;
}

// Kernel for 1D average pooling
__global__ void modular_avg_pool1d_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    const int kernel_size,
    const int stride,
    const int padding,
    const int input_length,
    const int output_length,
    const int batch_size,
    const int in_channels) {

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_elements = batch_size * in_channels * output_length;

    if (idx >= total_elements) return;

    const int o = idx % output_length;
    const int channel = (idx / output_length) % in_channels;
    const int batch = idx / (output_length * in_channels);

    const int input_batch_offset = batch * in_channels * input_length;
    const int input_channel_offset = channel * input_length;
    const int input_base = input_batch_offset + input_channel_offset;

    const int start_idx = o * stride - padding;

    output[idx] = compute_avg_pool1d(input, kernel_size, stride, padding, input_length, start_idx, input_base);
}

// Host function to launch the CUDA kernel
torch::Tensor modular_avg_pool1d_forward(
    const torch::Tensor &x,
    int kernel_size,
    int stride,
    int padding) {

    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(kernel_size > 0 && stride > 0 && padding >= 0, "Invalid kernel parameters");

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int input_length = x.size(2);
    const int output_length = (input_length + 2 * padding - kernel_size) / stride + 1;

    auto output = torch::empty({batch_size, in_channels, output_length}, x.options());

    const int total_elements = batch_size * in_channels * output_length;
    const int threads = 256;
    const int blocks = (total_elements + threads - 1) / threads;

    modular_avg_pool1d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        output.data_ptr<float>(),
        kernel_size,
        stride,
        padding,
        input_length,
        output_length,
        batch_size,
        in_channels
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &modular_avg_pool1d_forward, "Modular 1D Average Pooling forward (CUDA)");
}