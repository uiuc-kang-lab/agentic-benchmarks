#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>

namespace py = pybind11;

#define THREADS_PER_BLOCK 256
#define WARP_SIZE 32

template <typename scalar_t>
__global__ void depthwise_conv2d_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    int batch,
    int channels,
    int in_h, int in_w,
    int out_h, int out_w,
    int k,
    int stride,
    int padding,
    int dilation) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch * channels * out_h * out_w;
    if (index >= total)
        return;

    int ow = index % out_w;
    int tmp = index / out_w;
    int oh = tmp % out_h;
    tmp = tmp / out_h;
    int c = tmp % channels;
    int n = tmp / channels;

    scalar_t sum = 0;
    for (int i = 0; i < k; ++i) {
        for (int j = 0; j < k; ++j) {
            int ih = oh * stride - padding + i * dilation;
            int iw = ow * stride - padding + j * dilation;
            if (ih >= 0 && ih < in_h && iw >= 0 && iw < in_w) {
                int input_idx = n * channels * in_h * in_w + c * in_h * in_w + ih * in_w + iw;
                int weight_idx = c * k * k + i * k + j;
                sum += input[input_idx] * weight[weight_idx];
            }
        }
    }
    if (bias != nullptr)
        sum += bias[c];
    output[index] = sum;
}

template <typename scalar_t>
__global__ void pointwise_conv2d_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    int batch,
    int in_channels,
    int out_channels,
    int h,
    int w) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch * out_channels * h * w;
    if (index >= total)
        return;

    int ow = index % w;
    int tmp = index / w;
    int oh = tmp % h;
    tmp = tmp / h;
    int oc = tmp % out_channels;
    int n = tmp / out_channels;

    // Calculate initial sum for this thread's portion of input channels
    scalar_t sum = 0;
    int lane_id = threadIdx.x % WARP_SIZE;
    int warp_channels = (in_channels + WARP_SIZE - 1) / WARP_SIZE;
    
    // Each thread processes multiple channels
    for (int ic_base = 0; ic_base < in_channels; ic_base += WARP_SIZE) {
        int ic = ic_base + lane_id;
        if (ic < in_channels) {
            int input_idx = n * in_channels * h * w + ic * h * w + oh * w + ow;
            int weight_idx = oc * in_channels + ic;
            sum += input[input_idx] * weight[weight_idx];
        }
    }

    // Warp-level reduction using shuffle operations
    #pragma unroll
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    // Only the first thread in each warp writes the result
    if (lane_id == 0) {
        if (bias != nullptr)
            sum += bias[oc];
        output[index] = sum;
    }
}

torch::Tensor forward_cuda(
    const torch::Tensor& x,
    const torch::Tensor& depthwise_weight,
    const torch::Tensor& pointwise_weight,
    const torch::Tensor& depthwise_bias,
    const torch::Tensor& pointwise_bias,
    int stride,
    int padding,
    int dilation) {

    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
    TORCH_CHECK(depthwise_weight.is_cuda(), "Depthwise weight must be a CUDA tensor");
    TORCH_CHECK(pointwise_weight.is_cuda(), "Pointwise weight must be a CUDA tensor");

    int batch = x.size(0);
    int in_channels = x.size(1);
    int in_h = x.size(2);
    int in_w = x.size(3);
    int k = depthwise_weight.size(2);
    int out_h = (in_h + 2 * padding - dilation * (k - 1) - 1) / stride + 1;
    int out_w = (in_w + 2 * padding - dilation * (k - 1) - 1) / stride + 1;

    auto depthwise_output = torch::empty({batch, in_channels, out_h, out_w}, x.options());

    int total_depthwise = batch * in_channels * out_h * out_w;
    int threads = THREADS_PER_BLOCK;
    int blocks = (total_depthwise + threads - 1) / threads;

    const void* depthwise_bias_ptr = (depthwise_bias.defined() && depthwise_bias.numel() > 0)
                                        ? depthwise_bias.data_ptr()
                                        : nullptr;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "depthwise_conv2d_cuda", ([&] {
        depthwise_conv2d_kernel<scalar_t><<<blocks, threads>>>(
            x.data_ptr<scalar_t>(),
            depthwise_weight.data_ptr<scalar_t>(),
            reinterpret_cast<const scalar_t*>(depthwise_bias_ptr),
            depthwise_output.data_ptr<scalar_t>(),
            batch,
            in_channels,
            in_h, in_w,
            out_h, out_w,
            k,
            stride,
            padding,
            dilation);
    }));

    int out_channels = pointwise_weight.size(0);
    auto output = torch::empty({batch, out_channels, out_h, out_w}, x.options());
    int total_pointwise = batch * out_channels * out_h * out_w;
    blocks = (total_pointwise + threads - 1) / threads;

    const void* pointwise_bias_ptr = (pointwise_bias.defined() && pointwise_bias.numel() > 0)
                                        ? pointwise_bias.data_ptr()
                                        : nullptr;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "pointwise_conv2d_cuda", ([&] {
        pointwise_conv2d_kernel<scalar_t><<<blocks, threads>>>(
            depthwise_output.data_ptr<scalar_t>(),
            pointwise_weight.data_ptr<scalar_t>(),
            reinterpret_cast<const scalar_t*>(pointwise_bias_ptr),
            output.data_ptr<scalar_t>(),
            batch,
            in_channels,
            out_channels,
            out_h, out_w);
    }));

    return output;
}

at::Tensor toTensor(const py::object& obj) {
    if (obj.is_none()) {
        return at::Tensor();
    }
    try {
        return obj.cast<at::Tensor>();
    } catch (const py::cast_error& e) {
        if (py::hasattr(obj, "data")) {
            return obj.attr("data").cast<at::Tensor>();
        }
        throw std::runtime_error("Expected a torch Tensor or Parameter.");
    }
}

at::Tensor forward_wrapper(py::object x_obj,
                           py::object depthwise_weight_obj,
                           py::object pointwise_weight_obj,
                           py::object depthwise_bias_obj,
                           py::object pointwise_bias_obj,
                           int stride,
                           int padding,
                           int dilation) {

    auto x = toTensor(x_obj);
    auto depthwise_weight = toTensor(depthwise_weight_obj);
    auto pointwise_weight = toTensor(pointwise_weight_obj);
    auto depthwise_bias = toTensor(depthwise_bias_obj);
    auto pointwise_bias = toTensor(pointwise_bias_obj);

    return forward_cuda(x, depthwise_weight, pointwise_weight,
                       depthwise_bias, pointwise_bias,
                       stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_wrapper, "CUDA depthwise separable convolution forward");
}