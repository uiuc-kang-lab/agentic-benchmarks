#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__constant__ int d_C;
__constant__ int d_total_vectors;
__constant__ int d_stride_C;
__constant__ int d_outer_stride;

template <typename scalar_t>
__global__ void l2_normalize_kernel(
    const scalar_t* input,
    scalar_t* output) {

    const int vector_idx = blockIdx.x;
    if (vector_idx >= d_total_vectors) return;

    const int base_offset = vector_idx * d_outer_stride;

    scalar_t sum = 0.0;

    // Compute sum of squares
    for (int k = threadIdx.x; k < d_C; k += blockDim.x) {
        const scalar_t val = input[base_offset + k * d_stride_C];
        sum += val * val;
    }

    // Block-wise reduction
    __shared__ scalar_t shared_sum[256];
    scalar_t warp_sum = 0.0;
    const int lane = threadIdx.x % 32;
    const int wid = threadIdx.x / 32;

    for (int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    if (lane == 0) {
        shared_sum[wid] = sum;
    }
    __syncthreads();

    scalar_t block_sum = (threadIdx.x < blockDim.x / 32) ? shared_sum[lane] : 0.0;
    if (wid == 0) {
        for (int offset = 16; offset > 0; offset /= 2) {
            block_sum += __shfl_down_sync(0xffffffff, block_sum, offset);
        }
        if (lane == 0) {
            shared_sum[0] = block_sum;
        }
    }
    __syncthreads();

    const scalar_t inv_norm = 1.0 / (sqrt(shared_sum[0]) + 1e-12);

    // Normalize and write output
    for (int k = threadIdx.x; k < d_C; k += blockDim.x) {
        output[base_offset + k * d_stride_C] = input[base_offset + k * d_stride_C] * inv_norm;
    }
}

torch::Tensor forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(input.dim() >= 1, "Input must have at least 1 dimension");

    const int C = input.size(1);
    const int total_vectors = input.numel() / C;
    const int stride_C = input.stride(1);
    const int outer_stride = input.stride(0);

    // Copy constants to device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_C), &C, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_total_vectors), &total_vectors, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_stride_C), &stride_C, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_outer_stride), &outer_stride, sizeof(int));

    auto output = torch::empty_like(input);

    const int threads = 256;
    const int blocks = total_vectors;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "l2_normalize", [&] {
        l2_normalize_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>()
        );
    });

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "L2 normalization along dim=1");
}