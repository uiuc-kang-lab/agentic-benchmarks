#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32

// CUDA kernel optimized for memory coalescing
__global__ void coalesced_matmul_kernel(const float* __restrict__ A,
                                         const float* __restrict__ B,
                                         float* __restrict__ C,
                                         int N) {
    __shared__ float shared_A[TILE_SIZE][TILE_SIZE];
    __shared__ float shared_B[TILE_SIZE][TILE_SIZE];

    // Compute global row and column indices
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    // Loop over tiles
    for (int m = 0; m < (N + TILE_SIZE - 1) / TILE_SIZE; ++m) {
        // Load tile from A with coalesced access
        int a_col = m * TILE_SIZE + threadIdx.x;
        if (row < N && a_col < N)
            shared_A[threadIdx.y][threadIdx.x] = A[row * N + a_col];
        else
            shared_A[threadIdx.y][threadIdx.x] = 0.0f;

        // Load tile from B with coalesced access
        int b_row = m * TILE_SIZE + threadIdx.y;
        if (b_row < N && col < N)
            shared_B[threadIdx.y][threadIdx.x] = B[b_row * N + col];
        else
            shared_B[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Multiply the two tiles together
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += shared_A[threadIdx.y][k] * shared_B[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write the result back to global memory
    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

// C++ interface
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be of same size");

    int N = A.size(0);
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, A.get_device());
    auto C = torch::zeros({N, N}, options);

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    coalesced_matmul_kernel<<<gridDim, blockDim>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), N);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Coalesced Matrix Multiplication (CUDA)");
}
