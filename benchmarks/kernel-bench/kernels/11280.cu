#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void warp_aligned_cosine_loss_kernel(const float* __restrict__ predictions,
                                               const float* __restrict__ targets,
                                               float* output,
                                               int N,
                                               int D) {
    extern __shared__ float s_data[];
    const int warp_size = 32;
    const int warps_per_block = blockDim.x / warp_size;
    const int warp_id = threadIdx.x / warp_size;
    const int lane_id = threadIdx.x % warp_size;
    const int row = blockIdx.x;
    
    // Align data pointers for the current row
    const float* pred_row = predictions + row * D;
    const float* target_row = targets + row * D;
    
    // Initialize accumulators
    float sum_dot = 0.0f;
    float sum_pred_sq = 0.0f;
    float sum_target_sq = 0.0f;
    
    // Vectorized load and compute
    #pragma unroll 4
    for (int i = threadIdx.x; i < D; i += blockDim.x) {
        float p = pred_row[i];
        float t = target_row[i];
        sum_dot += p * t;
        sum_pred_sq += p * p;
        sum_target_sq += t * t;
    }
    
    // Warp-level reduction using shuffle operations
    #pragma unroll
    for (int offset = warp_size/2; offset > 0; offset >>= 1) {
        sum_dot += __shfl_down_sync(0xffffffff, sum_dot, offset);
        sum_pred_sq += __shfl_down_sync(0xffffffff, sum_pred_sq, offset);
        sum_target_sq += __shfl_down_sync(0xffffffff, sum_target_sq, offset);
    }
    
    // First thread in each warp writes to shared memory
    if (lane_id == 0) {
        s_data[warp_id] = sum_dot;
        s_data[warp_id + warps_per_block] = sum_pred_sq;
        s_data[warp_id + 2 * warps_per_block] = sum_target_sq;
    }
    __syncthreads();
    
    // Final reduction using the first warp
    if (warp_id == 0) {
        sum_dot = (lane_id < warps_per_block) ? s_data[lane_id] : 0.0f;
        sum_pred_sq = (lane_id < warps_per_block) ? s_data[lane_id + warps_per_block] : 0.0f;
        sum_target_sq = (lane_id < warps_per_block) ? s_data[lane_id + 2 * warps_per_block] : 0.0f;
        
        // Warp-level reduction for final values
        #pragma unroll
        for (int offset = warp_size/2; offset > 0; offset >>= 1) {
            sum_dot += __shfl_down_sync(0xffffffff, sum_dot, offset);
            sum_pred_sq += __shfl_down_sync(0xffffffff, sum_pred_sq, offset);
            sum_target_sq += __shfl_down_sync(0xffffffff, sum_target_sq, offset);
        }
        
        if (lane_id == 0) {
            const float eps = 1e-8f;
            float norm_pred = sqrtf(sum_pred_sq);
            float norm_target = sqrtf(sum_target_sq);
            float denominator = norm_pred * norm_target;
            denominator = fmaxf(denominator, eps);
            float cos_sim = sum_dot / denominator;
            atomicAdd(output, (1.0f - cos_sim) / N);
        }
    }
}

torch::Tensor warp_aligned_cosine_loss_forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.dim() == 2, "predictions must be 2D");
    TORCH_CHECK(targets.dim() == 2, "targets must be 2D");
    TORCH_CHECK(predictions.sizes() == targets.sizes(), "Input tensors must have the same shape");
    TORCH_CHECK(predictions.scalar_type() == torch::kFloat32, "predictions must be float32");
    TORCH_CHECK(targets.scalar_type() == torch::kFloat32, "targets must be float32");

    int N = predictions.size(0);
    int D = predictions.size(1);

    auto output = torch::zeros({1}, predictions.options());
    
    // Ensure block size is multiple of warp size for aligned access
    const int block_size = 128; // 4 warps per block
    const int warps_per_block = block_size / 32;
    size_t shared_mem = 3 * warps_per_block * sizeof(float);

    warp_aligned_cosine_loss_kernel<<<N, block_size, shared_mem>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        N,
        D
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &warp_aligned_cosine_loss_forward, "Warp Aligned Cosine Similarity Loss Forward (CUDA)");
}