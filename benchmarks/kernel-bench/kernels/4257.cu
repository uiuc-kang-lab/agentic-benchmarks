#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")

// Warp-level reduction using shuffle intrinsics
__inline__ __device__ float warpReduceSum(float val) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xffffffff, val, offset);
    return val;
}

// Compute partial sums for one channel
__device__ void computePartialSums(const float* __restrict__ input,
                                     int c, int N, int C, int H, int W,
                                     int tid, int stride,
                                     float &partialSum, float &partialSumSq) {
    int numElements = N * H * W;
    partialSum = 0.0f;
    partialSumSq = 0.0f;
    for (int i = tid; i < numElements; i += stride) {
        int n = i / (H * W);
        int rem = i % (H * W);
        int h = rem / W;
        int w = rem % W;
        int idx = ((n * C + c) * H + h) * W + w;
        float val = input[idx];
        partialSum += val;
        partialSumSq += val * val;
    }
}

// Block-level reduction using warp shuffle and shared memory
__device__ void blockReduceSum(float &sum, float &sumSq) {
    int lane = threadIdx.x % warpSize;
    int warpId = threadIdx.x / warpSize;
    
    float sum_val = warpReduceSum(sum);
    float sumSq_val = warpReduceSum(sumSq);
    
    __shared__ float sharedSum[32]; // max 32 warps per block
    __shared__ float sharedSumSq[32];
    
    if (lane == 0) {
        sharedSum[warpId] = sum_val;
        sharedSumSq[warpId] = sumSq_val;
    }
    __syncthreads();

    // Final reduction by first warp
    if (threadIdx.x < warpSize) {
        float finalSum = (threadIdx.x < (blockDim.x + warpSize - 1) / warpSize) ? sharedSum[threadIdx.x] : 0.0f;
        float finalSumSq = (threadIdx.x < (blockDim.x + warpSize - 1) / warpSize) ? sharedSumSq[threadIdx.x] : 0.0f;
        finalSum = warpReduceSum(finalSum);
        finalSumSq = warpReduceSum(finalSumSq);
        if (threadIdx.x == 0) {
            sum = finalSum;
            sumSq = finalSumSq;
        }
    }
    __syncthreads();
}

// Normalize a value given the mean, inverse standard deviation, weight and bias
__device__ inline float normalizeValue(float val, float mean, float invStd, float w, float b) {
    return (val - mean) * invStd * w + b;
}

// Kernel that performs BatchNorm for one channel using tunable block size
__global__ void tunable_blocksize_batch_norm_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ running_mean,
    float* __restrict__ running_var,
    bool training,
    float momentum,
    float eps,
    float* __restrict__ output,
    int N,
    int C,
    int H,
    int W) {

    // Each block processes one channel
    int c = blockIdx.x;
    int tid = threadIdx.x;
    int stride = blockDim.x;
    int numElements = N * H * W;

    float partialSum, partialSumSq;
    computePartialSums(input, c, N, C, H, W, tid, stride, partialSum, partialSumSq);
    blockReduceSum(partialSum, partialSumSq);

    __shared__ float stats[2]; // stats[0] = mean, stats[1] = variance
    float mean, var;
    if (tid == 0) {
        mean = partialSum / numElements;
        var = partialSumSq / numElements - mean * mean;
        if (training) {
            running_mean[c] = (1.0f - momentum) * running_mean[c] + momentum * mean;
            running_var[c] = (1.0f - momentum) * running_var[c] + momentum * var;
        } else {
            mean = running_mean[c];
            var = running_var[c];
        }
        stats[0] = mean;
        stats[1] = var;
    }
    __syncthreads();
    mean = stats[0];
    var = stats[1];

    float invStd = rsqrtf(var + eps);
    float channelWeight = weight[c];
    float channelBias = bias[c];

    // Normalize: each thread processes a subset of elements
    for (int i = tid; i < numElements; i += stride) {
        int n = i / (H * W);
        int rem = i % (H * W);
        int h = rem / W;
        int w = rem % W;
        int idx = ((n * C + c) * H + h) * W + w;
        float val = input[idx];
        output[idx] = normalizeValue(val, mean, invStd, channelWeight, channelBias);
    }
}

// Host function: selects optimal block size from a set and launches the kernel
torch::Tensor tunable_forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    torch::Tensor running_mean,
    torch::Tensor running_var,
    bool training,
    float momentum,
    float eps) {

    CHECK_CUDA(input);
    CHECK_CUDA(weight);
    CHECK_CUDA(bias);
    CHECK_CUDA(running_mean);
    CHECK_CUDA(running_var);

    CHECK_CONTIGUOUS(input);
    CHECK_CONTIGUOUS(weight);
    CHECK_CONTIGUOUS(bias);
    CHECK_CONTIGUOUS(running_mean);
    CHECK_CONTIGUOUS(running_var);

    int N = input.size(0);
    int C = input.size(1);
    int H = input.size(2);
    int W = input.size(3);
    int numElements = N * H * W;

    // Heuristic to select block size based on the number of elements per channel
    int block_size;
    if (numElements < 1024) {
        block_size = 32;
    } else if (numElements < 4096) {
        block_size = 64;
    } else if (numElements < 16384) {
        block_size = 128;
    } else if (numElements < 65536) {
        block_size = 256;
    } else {
        block_size = 512;
    }

    auto output = torch::empty_like(input);

    // Each channel is processed by one block
    dim3 grid(C);
    // Launch kernel with computed block size
    tunable_blocksize_batch_norm_kernel<<<grid, block_size>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.data_ptr<float>(),
        running_mean.data_ptr<float>(),
        running_var.data_ptr<float>(),
        training,
        momentum,
        eps,
        output.data_ptr<float>(),
        N, C, H, W
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &tunable_forward_cuda, "Tunable Block Size BatchNorm forward (CUDA)");
}
