#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

__global__ void kldiv_vectorized_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets,
    float* __restrict__ output,
    const int64_t n) {

    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x * 4;
    
    float4 sum = {0, 0, 0, 0};
    
    // Vectorized memory access with float4
    for (int64_t i = tid * 4; i < n; i += stride) {
        float4 log_pred = *reinterpret_cast<const float4*>(&log_predictions[i]);
        float4 target = *reinterpret_cast<const float4*>(&targets[i]);
        
        sum.x += expf(log_pred.x) - target.x * log_pred.x;
        sum.y += expf(log_pred.y) - target.y * log_pred.y;
        sum.z += expf(log_pred.z) - target.z * log_pred.z;
        sum.w += expf(log_pred.w) - target.w * log_pred.w;
    }

    // Horizontal sum within thread
    float thread_sum = sum.x + sum.y + sum.z + sum.w;

    // Warp-level reduction
    for (int offset = 16; offset > 0; offset /= 2)
        thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);

    // Block-level reduction
    __shared__ float block_sum[32];
    if (threadIdx.x % 32 == 0)
        block_sum[threadIdx.x / 32] = thread_sum;
    __syncthreads();

    // Final reduction and atomic add
    if (threadIdx.x == 0) {
        float total = 0;
        for (int i = 0; i < blockDim.x / 32; i++)
            total += block_sum[i];
        atomicAdd(output, total);
    }
}

torch::Tensor kl_div_cuda_forward(
    const torch::Tensor& log_predictions,
    const torch::Tensor& targets) {
    
    const int64_t n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());

    // H100-optimized launch config
    const int threads = 256;
    const int blocks = 144 * 4;  // 144 SMs * 4 waves

    // Handle non-divisible sizes
    const int64_t padded_n = (n + 3) & ~3;
    kldiv_vectorized_kernel<<<blocks, threads>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        padded_n
    );

    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence vectorized forward");
}