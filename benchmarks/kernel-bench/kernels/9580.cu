#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#define NUM_STREAMS 4

template <typename scalar_t>
__global__ void depthwiseConv2DKernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ w,
    const scalar_t* __restrict__ b,
    scalar_t* __restrict__ out,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int kernel_size,
    const int out_height,
    const int out_width,
    const int stride,
    const int padding,
    const int batch_offset)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch_size * in_channels * out_height * out_width;
    if (idx >= total) return;

    int w_out_idx = idx % out_width;
    int tmp = idx / out_width;
    int h_out_idx = tmp % out_height;
    tmp /= out_height;
    int c = tmp % in_channels;
    int n = (tmp / in_channels) + batch_offset;

    scalar_t value = 0;
    #pragma unroll
    for (int kh = 0; kh < kernel_size; kh++) {
        #pragma unroll
        for (int kw = 0; kw < kernel_size; kw++) {
            int h_in = h_out_idx * stride - padding + kh;
            int w_in = w_out_idx * stride - padding + kw;
            if (h_in >= 0 && h_in < in_height && w_in >= 0 && w_in < in_width) {
                int x_index = ((n * in_channels + c) * in_height + h_in) * in_width + w_in;
                int w_index = ((c * 1 + 0) * kernel_size + kh) * kernel_size + kw;
                value += x[x_index] * w[w_index];
            }
        }
    }
    value += b[c];
    out[idx] = value;
}

torch::Tensor forward_impl(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int groups)
{
    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);
    const int kernel_size = weight.size(2);
    const int out_height = (in_height + 2 * padding - kernel_size) / stride + 1;
    const int out_width = (in_width + 2 * padding - kernel_size) / stride + 1;

    auto out = torch::empty({batch_size, in_channels, out_height, out_width}, x.options());

    hipStream_t streams[NUM_STREAMS];
    for(int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    const int batch_per_stream = (batch_size + NUM_STREAMS - 1) / NUM_STREAMS;
    const int threads = 256;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "depthwise_conv2d_forward", ([&] {
        for(int i = 0; i < NUM_STREAMS; i++) {
            int current_batch_start = i * batch_per_stream;
            int current_batch_size = std::min(batch_per_stream, 
                                            batch_size - current_batch_start);
            if(current_batch_size <= 0) continue;

            const int total = current_batch_size * in_channels * out_height * out_width;
            const int blocks = (total + threads - 1) / threads;

            depthwiseConv2DKernel<scalar_t><<<blocks, threads, 0, streams[i]>>>(
                x.data_ptr<scalar_t>(),
                weight.data_ptr<scalar_t>(),
                bias.data_ptr<scalar_t>(),
                out.data_ptr<scalar_t>(),
                current_batch_size, in_channels, in_height, in_width,
                kernel_size, out_height, out_width,
                stride, padding, current_batch_start
            );
        }
    }));

    // Synchronize all streams
    for(int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    return out;
}

namespace py = pybind11;

torch::Tensor forward_wrap(
    torch::Tensor x,
    torch::Tensor weight,
    py::object bias_obj,
    int stride,
    int padding,
    int groups)
{
    torch::Tensor bias;
    if (bias_obj.is_none()) {
        bias = torch::zeros({x.size(1)}, x.options());
    } else {
        bias = bias_obj.cast<torch::Tensor>();
    }
    return forward_impl(x, weight, bias, stride, padding, groups);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        &forward_wrap,
        "Depthwise conv2d forward (handles optional bias)",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = 1,
        py::arg("padding") = 0,
        py::arg("groups") = 1
    );
}