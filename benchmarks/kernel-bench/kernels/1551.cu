#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

__global__ void upper_triangular_matmul_kernel(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N && row <= col) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            if (k >= row && k <= col) {
                sum += A[row * N + k] * B[k * N + col];
            }
        }
        C[row * N + col] = sum;
    }
}

torch::Tensor upper_triangular_matmul(torch::Tensor A, torch::Tensor B) {
    int N = A.size(0);

    auto C = torch::zeros_like(A);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    upper_triangular_matmul_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &upper_triangular_matmul, "Upper triangular matrix multiplication");
}