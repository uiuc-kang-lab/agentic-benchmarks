#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <vector>

namespace py = pybind11;

// Use constant memory for kernel weights (max 64KB)
__constant__ float c_weight[16384];

// Combined kernel: each warp computes one output element using warp-level reduction
// This kernel fuses dynamic block configuration ideas with efficient warp shuffle reduction,
// eliminating extra shared memory usage while handling various output sizes.

__global__ void conv_transpose2d_forward_kernel_combined(
    const float* __restrict__ input,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int N,
    const int C_in,
    const int H_in,
    const int W_in,
    const int C_out,
    const int H_out,
    const int W_out,
    const int kH,
    const int kW,
    const int sH,
    const int sW,
    const int pH,
    const int pW
) {
    // Each warp computes one output element.
    int num_warps_per_block = blockDim.x / warpSize;  // warpSize is typically 32
    int warp_id_in_block = threadIdx.x / warpSize;
    int global_warp_id = blockIdx.x * num_warps_per_block + warp_id_in_block;
    int total_output = N * C_out * H_out * W_out;

    if (global_warp_id >= total_output) return;

    // Decode the output index from the global warp id
    int ow = global_warp_id % W_out;
    int oh = (global_warp_id / W_out) % H_out;
    int oc = (global_warp_id / (W_out * H_out)) % C_out;
    int n  = global_warp_id / (W_out * H_out * C_out);

    float local_sum = 0.0f;
    int total_work = C_in * kH * kW;

    int lane = threadIdx.x % warpSize;  // lane within the warp

    // Each lane sums over a portion of the reduction dimension
    for (int i = lane; i < total_work; i += warpSize) {
        int ic = i / (kH * kW);
        int rem = i % (kH * kW);
        int kh = rem / kW;
        int kw = rem % kW;

        int i_val = oh + pH - kh;
        int j_val = ow + pW - kw;
        
        if ((i_val % sH == 0) && (j_val % sW == 0)) {
            int i_in = i_val / sH;
            int j_in = j_val / sW;
            if (i_in >= 0 && i_in < H_in && j_in >= 0 && j_in < W_in) {
                int input_idx = ((n * C_in + ic) * H_in + i_in) * W_in + j_in;
                int weight_idx = ((ic * C_out + oc) * kH + kh) * kW + kw;
                local_sum += input[input_idx] * c_weight[weight_idx];
            }
        }
    }

    // Perform warp-level reduction using shuffle intrinsics
    unsigned int mask = 0xffffffff;
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        local_sum += __shfl_down_sync(mask, local_sum, offset);
    }

    // The first thread in the warp writes the final result
    if (lane == 0) {
        if (bias != nullptr) {
            local_sum += bias[oc];
        }
        int output_idx = ((n * C_out + oc) * H_out + oh) * W_out + ow;
        output[output_idx] = local_sum;
    }
}

// Host function: sets up and launches the combined kernel

torch::Tensor conv_transpose2d_forward(
    torch::Tensor x,
    torch::Tensor weight,
    py::object bias_obj,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding
) {
    int weight_size = weight.numel() * sizeof(float);
    const int max_const_size = 64 * 1024;
    if (weight_size > max_const_size) {
        // Fallback to cuDNN for large weights
        c10::optional<torch::Tensor> bias = c10::nullopt;
        if (!bias_obj.is_none()) {
            bias = bias_obj.cast<torch::Tensor>();
        }
        return at::conv_transpose2d(x, weight, bias, stride, padding);
    }

    // Copy weight to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(c_weight), weight.data_ptr<float>(), weight_size);

    torch::Tensor bias;
    const float* bias_ptr = nullptr;
    if (!bias_obj.is_none()) {
        bias = bias_obj.cast<torch::Tensor>();
        bias_ptr = bias.data_ptr<float>();
    }

    const int N = x.size(0);
    const int C_in = x.size(1);
    const int H_in = x.size(2);
    const int W_in = x.size(3);
    const int C_out = weight.size(1);
    const int kH = weight.size(2);
    const int kW = weight.size(3);
    const int sH = stride[0];
    const int sW = stride[1];
    const int pH = padding[0];
    const int pW = padding[1];

    const int H_out = (H_in - 1) * sH - 2 * pH + kH;
    const int W_out = (W_in - 1) * sW - 2 * pW + kW;

    auto output = torch::zeros({N, C_out, H_out, W_out}, x.options());

    // Launch configuration: assign one warp per output element
    const int BLOCK_SIZE = 256; // must be a multiple of warpSize (32)
    int warpsPerBlock = BLOCK_SIZE / 32;
    int total_output = N * C_out * H_out * W_out;
    int grid = (total_output + warpsPerBlock - 1) / warpsPerBlock;

    conv_transpose2d_forward_kernel_combined<<<grid, BLOCK_SIZE>>>(
         x.data_ptr<float>(),
         bias_ptr,
         output.data_ptr<float>(),
         N, C_in, H_in, W_in,
         C_out, H_out, W_out,
         kH, kW, sH, sW, pH, pW
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward, "Combined Conv Transpose 2D forward kernel with warp-level reduction",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride"),
          py::arg("padding"));
}
