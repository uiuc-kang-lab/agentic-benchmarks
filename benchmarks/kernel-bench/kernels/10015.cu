#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Tunable parameters
#define BLOCK_SIZE 16
#define ELEMENTS_PER_THREAD 4

// Combined CUDA kernel with improved global memory access using __ldg

__global__ void optimized_depthwise_conv2d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int in_h,
    int in_w,
    int out_channels,
    int out_h,
    int out_w,
    int kernel_h,
    int kernel_w,
    int stride_h,
    int stride_w,
    int padding_h,
    int padding_w,
    int dilation_h,
    int dilation_w,
    int groups,
    int channels_per_group
) {
    int b = blockIdx.z / out_channels;
    int c = blockIdx.z % out_channels;
    int g = c / channels_per_group;
    int m = c % channels_per_group;

    const int tile_out_width = BLOCK_SIZE * ELEMENTS_PER_THREAD;
    const int tile_out_height = BLOCK_SIZE;

    int shared_tile_width = tile_out_width * stride_w + (kernel_w - 1) * dilation_w;
    int shared_tile_height = tile_out_height * stride_h + (kernel_h - 1) * dilation_h;
    int base_in_x = blockIdx.x * tile_out_width * stride_w - padding_w;
    int base_in_y = blockIdx.y * tile_out_height * stride_h - padding_h;

    extern __shared__ float shared_input[];
    int shared_size = shared_tile_width * shared_tile_height;

    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int thread_id = tidy * blockDim.x + tidx;
    int total_threads = blockDim.x * blockDim.y;

    for (int idx = thread_id; idx < shared_size; idx += total_threads) {
        int sh_y = idx / shared_tile_width;
        int sh_x = idx % shared_tile_width;
        int in_y = base_in_y + sh_y;
        int in_x = base_in_x + sh_x;
        if (in_y >= 0 && in_y < in_h && in_x >= 0 && in_x < in_w) {
            shared_input[sh_y * shared_tile_width + sh_x] = __ldg(
                &input[((b * in_channels + g) * in_h + in_y) * in_w + in_x]
            );
        } else {
            shared_input[sh_y * shared_tile_width + sh_x] = 0.0f;
        }
    }
    __syncthreads();

    int out_y = blockIdx.y * tile_out_height + tidy;
    if (out_y < out_h) {
        int base_out_x = blockIdx.x * tile_out_width + tidx * ELEMENTS_PER_THREAD;
        float results[ELEMENTS_PER_THREAD] = {0.f, 0.f, 0.f, 0.f};

        for (int kh = 0; kh < kernel_h; kh++) {
            int sh_y = tidy * stride_h + kh * dilation_h;
            for (int kw = 0; kw < kernel_w; kw++) {
                float w_val = __ldg(&weight[((g * channels_per_group + m) * kernel_h + kh) * kernel_w + kw]);
                int base_sh_x = tidx * ELEMENTS_PER_THREAD * stride_w + kw * dilation_w;
                #pragma unroll
                for (int j = 0; j < ELEMENTS_PER_THREAD; j++) {
                    int sh_x = base_sh_x + j * stride_w;
                    float in_val = shared_input[sh_y * shared_tile_width + sh_x];
                    results[j] += in_val * w_val;
                }
            }
        }

        #pragma unroll
        for (int j = 0; j < ELEMENTS_PER_THREAD; j++) {
            int out_x = base_out_x + j;
            if (out_x < out_w) {
                float res = results[j];
                if (bias != nullptr) {
                    res += __ldg(&bias[c]);
                }
                int out_idx = ((b * out_channels + c) * out_h + out_y) * out_w + out_x;
                output[out_idx] = res;
            }
        }
    }
}

// Host function with kernel launch setup

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    int stride_h,
    int stride_w,
    int padding_h,
    int padding_w,
    int dilation_h,
    int dilation_w,
    int groups
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int in_h = x.size(2);
    int in_w = x.size(3);

    int kernel_h = weight.size(2);
    int kernel_w = weight.size(3);
    int out_channels = groups * weight.size(1);
    int channels_per_group = out_channels / groups;

    int out_h = (in_h + 2 * padding_h - dilation_h * (kernel_h - 1) - 1) / stride_h + 1;
    int out_w = (in_w + 2 * padding_w - dilation_w * (kernel_w - 1) - 1) / stride_w + 1;

    auto output = torch::empty({batch_size, out_channels, out_h, out_w}, x.options());

    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        bias_ptr = bias->data_ptr<float>();
    }

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(
        (out_w + (BLOCK_SIZE * ELEMENTS_PER_THREAD) - 1) / (BLOCK_SIZE * ELEMENTS_PER_THREAD),
        (out_h + BLOCK_SIZE - 1) / BLOCK_SIZE,
        batch_size * out_channels
    );

    int tile_out_width = BLOCK_SIZE * ELEMENTS_PER_THREAD;
    int tile_out_height = BLOCK_SIZE;
    int shared_tile_width = tile_out_width * stride_w + (kernel_w - 1) * dilation_w;
    int shared_tile_height = tile_out_height * stride_h + (kernel_h - 1) * dilation_h;
    int shared_mem_size = shared_tile_width * shared_tile_height * sizeof(float);

    optimized_depthwise_conv2d_kernel<<<blocks, threads, shared_mem_size>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        in_h,
        in_w,
        out_channels,
        out_h,
        out_w,
        kernel_h,
        kernel_w,
        stride_h,
        stride_w,
        padding_h,
        padding_w,
        dilation_h,
        dilation_w,
        groups,
        channels_per_group
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Depthwise Conv2D forward (CUDA) with global memory __ldg");
}