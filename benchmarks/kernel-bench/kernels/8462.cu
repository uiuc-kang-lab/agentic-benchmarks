#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <hip/hip_runtime.h>

__device__ float warpReduceSum(float val) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__global__ void conv_transpose2d_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    float* __restrict__ output,
    int x_height, int x_width,
    int weight_height, int weight_width,
    int out_height, int out_width,
    int stride_h, int stride_w,
    int pad_h, int pad_w) {

    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (out_x < out_width && out_y < out_height) {
        float sum = 0.0f;
        
        // Calculate the input position
        int in_x = (out_x + pad_w) / stride_w;
        int in_y = (out_y + pad_h) / stride_h;
        
        // Check if the input position is valid and if we're at a valid stride position
        if (in_x >= 0 && in_x < x_width && in_y >= 0 && in_y < x_height &&
            (out_x + pad_w) % stride_w == 0 && (out_y + pad_h) % stride_h == 0) {
            
            // For transposed convolution, we use the input value and multiply it with the entire kernel
            float input_val = x[in_y * x_width + in_x];
            
            // Calculate the kernel position
            int k_x = weight_width - 1 - ((out_x + pad_w) % (stride_w * weight_width) / stride_w);
            int k_y = weight_height - 1 - ((out_y + pad_h) % (stride_h * weight_height) / stride_h);
            
            if (k_x >= 0 && k_x < weight_width && k_y >= 0 && k_y < weight_height) {
                sum = input_val * weight[k_y * weight_width + k_x];
            }
        }
        
        output[out_y * out_width + out_x] = sum;
    }
}

void conv_transpose2d_cuda(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor output,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding) {

    const int x_height = x.size(0);
    const int x_width = x.size(1);
    const int weight_height = weight.size(0);
    const int weight_width = weight.size(1);
    const int out_height = output.size(0);
    const int out_width = output.size(1);

    const dim3 threadsPerBlock(32, 32);
    const dim3 numBlocks((out_width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                         (out_height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    conv_transpose2d_kernel<<<numBlocks, threadsPerBlock>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        output.data_ptr<float>(),
        x_height, x_width,
        weight_height, weight_width,
        out_height, out_width,
        stride[0], stride[1],
        padding[0], padding[1]);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_cuda, "ConvTranspose2D forward (CUDA)");
}