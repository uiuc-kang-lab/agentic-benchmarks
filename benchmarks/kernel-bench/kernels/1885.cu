#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32
#define CHUNK_SIZE 8

// Constant memory for frequently accessed parameters
__constant__ int d_N;
__constant__ int d_num_chunks;
__constant__ int d_chunk_sizes[256];  // For storing chunk size information

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C) {
    __shared__ float s_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float s_B[BLOCK_SIZE][BLOCK_SIZE];
    
    const int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    const int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    
    // Register array for accumulation
    float reg_C[CHUNK_SIZE] = {0.0f};
    
    // Calculate number of iterations based on constant memory value
    const int num_iterations = (row / BLOCK_SIZE) + 1;
    
    // Loop over block-level tiles
    for (int bk = 0; bk < num_iterations; bk++) {
        const int block_start = bk * BLOCK_SIZE;
        
        // Collaborative loading with vectorized memory access
        if (row < d_N && (block_start + threadIdx.x) < d_N) {
            s_A[threadIdx.y][threadIdx.x] = A[row * d_N + block_start + threadIdx.x];
        } else {
            s_A[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        if ((block_start + threadIdx.y) < d_N && col < d_N) {
            s_B[threadIdx.y][threadIdx.x] = B[(block_start + threadIdx.y) * d_N + col];
        } else {
            s_B[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        __syncthreads();
        
        // Register-level tiling for computation
        if (row < d_N && col < d_N && row >= col) {
            const int k_start = max(block_start, col);
            const int k_end = min(block_start + BLOCK_SIZE, row + 1);
            
            #pragma unroll
            for (int k = k_start; k < k_end; k += CHUNK_SIZE) {
                #pragma unroll
                for (int c = 0; c < CHUNK_SIZE && (k + c) < k_end; c++) {
                    reg_C[c] += s_A[threadIdx.y][k - block_start + c] * 
                               s_B[k - block_start + c][threadIdx.x];
                }
            }
        }
        
        __syncthreads();
    }
    
    // Reduction and writing results
    if (row < d_N && col < d_N) {
        if (row >= col) {
            float sum = 0.0f;
            #pragma unroll
            for (int i = 0; i < CHUNK_SIZE; i++) {
                sum += reg_C[i];
            }
            C[row * d_N + col] = sum;
        } else {
            C[row * d_N + col] = 0.0f;
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    const int N = A.size(0);
    auto C = torch::empty_like(A);

    // Copy constants to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_N), &N, sizeof(int));
    
    // Calculate and store chunk sizes in constant memory
    int num_chunks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_chunks), &num_chunks, sizeof(int));
    
    int chunk_sizes[256];  // Assuming max 256 chunks
    for (int i = 0; i < num_chunks; i++) {
        chunk_sizes[i] = min(BLOCK_SIZE, N - i * BLOCK_SIZE);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(d_chunk_sizes), chunk_sizes, num_chunks * sizeof(int));

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
                   (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>()
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}