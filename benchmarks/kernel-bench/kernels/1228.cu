#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

// Kernel that ensures minimal warp divergence
__global__ void einsum_kernel_minimized_divergence(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int BATCH, int I, int J, int L, int K
) {
    int index = blockIdx.x;
    int b = index / (I * J);
    int rem = index % (I * J);
    int i = rem / J;
    int j = rem % J;

    // Allocate shared memory for A[b,i,j,:]
    extern __shared__ float shared_A[];

    // Load A[b,i,j,:] into shared memory with coalesced memory access
    for (int l = threadIdx.x; l < L; l += blockDim.x) {
        int a_index = b * (I * J * L) + i * (J * L) + j * L + l;
        shared_A[l] = A[a_index];
    }
    __syncthreads();

    // Compute elements C[b,i,j,:] with minimized warp divergence
    for (int k = threadIdx.x; k < K; k += blockDim.x) {
        float sum = 0.0f;
        for (int l = 0; l < L; ++l) {
            // Access B[l*k:k] and shared_A[l] uniformly across threads
            sum += shared_A[l] * B[l * K + k];
        }
        C[b * (I * J * K) + i * (J * K) + j * K + k] = sum;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(A.dim() == 4, "Tensor A must be 4D");
    TORCH_CHECK(B.dim() == 2, "Tensor B must be 2D");
    TORCH_CHECK(A.size(3) == B.size(0), "Dimension mismatch: A.size(3) must equal B.size(0)");

    int BATCH = A.size(0);
    int I = A.size(1);
    int J = A.size(2);
    int L = A.size(3);
    int K = B.size(1);

    auto C = torch::zeros({BATCH, I, J, K}, A.options());

    int numBlocks = BATCH * I * J;
    int threads = 256;
    size_t sharedMemBytes = L * sizeof(float);

    einsum_kernel_minimized_divergence<<<numBlocks, threads, sharedMemBytes>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        BATCH, I, J, L, K
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Minimize warp divergence in 4D tensor-matrix multiplication (CUDA)");
}
