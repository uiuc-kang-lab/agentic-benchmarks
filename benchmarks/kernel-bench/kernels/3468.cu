#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Explicit specializations of gelu_function for float and double.
template <typename scalar_t>
__device__ inline scalar_t gelu_function(scalar_t x);

template <>
__device__ inline float gelu_function<float>(float x) {
    // GELU(x) = x * 0.5 * (1 + erf(x / sqrt(2)))
    return x * 0.5f * (1.0f + erff(x / 1.4142135623730951f));
}

template <>
__device__ inline double gelu_function<double>(double x) {
    return x * 0.5 * (1.0 + erf(x / 1.4142135623730951));
}

// CUDA kernel that applies the GELU activation element-wise with minimized warp divergence.
template <typename scalar_t>
__global__ void gelu_kernel_no_divergence(const scalar_t* __restrict__ x,
                                          scalar_t* __restrict__ y,
                                          size_t numel) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    scalar_t val = 0;
    if (index < numel) {
        val = x[index];
    }
    scalar_t result = gelu_function<scalar_t>(val);
    if (index < numel) {
        y[index] = result;
    }
}

// Forward function callable from Python.
torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
    
    auto output = torch::empty_like(x);
    size_t numel = x.numel();

    int threads = 256;
    int blocks = (numel + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "gelu_cuda_no_divergence", ([&] {
        gelu_kernel_no_divergence<scalar_t><<<blocks, threads>>>(x.data_ptr<scalar_t>(),
                                                                output.data_ptr<scalar_t>(),
                                                                numel);
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed : ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "GELU activation forward (CUDA)");
}