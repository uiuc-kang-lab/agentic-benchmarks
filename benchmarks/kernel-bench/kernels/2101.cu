#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel to compute C = tril(A * B) for lower triangular matrices A and B using warp-level primitives.
__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                       const float* __restrict__ B,
                                       float* __restrict__ C,
                                       int N) {
    // Compute the row and column indices for this thread.
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        // Only compute values for the lower triangular part.
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            float sum = 0.f;
            // For lower triangular matrices, A[i,k] is nonzero only if k <= i
            // and B[k,j] is nonzero only if j <= k.
            // Hence, only indices k from col to row contribute.
            for (int k = col; k <= row; ++k) {
                sum += A[row * N + k] * B[k * N + col];
            }

            // Use warp-level reduction to sum the values within a warp
            for (int offset = warpSize / 2; offset > 0; offset /= 2) {
                sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
            }

            // Write the result from the first thread of each warp
            if (threadIdx.x % warpSize == 0) {
                C[row * N + col] = sum;
            }
        }
    }
}

// C++ interface exposed to PyTorch.
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Define block and grid dimensions.
    const int threads = 32;  // Using a higher thread count per block for better warp utilization.
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threads - 1) / threads, (N + threads - 1) / threads);

    // Launch the CUDA kernel.
    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    // Check for kernel launch errors.
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}
