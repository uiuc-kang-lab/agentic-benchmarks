#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel using warp-level primitives instead of shared memory
__global__ void diag_matmul_warp_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    const int64_t N,
    const int64_t M
) {
    int row = blockIdx.x;
    const unsigned FULL_MASK = 0xffffffff;
    
    // Load diagonal element only in first thread of each warp
    float a_val = 0.0f;
    if (threadIdx.x % 32 == 0) {
        a_val = A[row];
    }
    // Broadcast the value to all threads in the warp
    a_val = __shfl_sync(FULL_MASK, a_val, 0);
    
    // Process four elements at a time when possible
    int col = threadIdx.x;
    const int stride = blockDim.x;
    const float4* B_vec = reinterpret_cast<const float4*>(B + row * M);
    float4* C_vec = reinterpret_cast<float4*>(C + row * M);
    
    // Handle vectorized loads first
    const int vec_limit = M / 4;
    while (col < vec_limit) {
        float4 b4 = B_vec[col];
        float4 c4;
        c4.x = a_val * b4.x;
        c4.y = a_val * b4.y;
        c4.z = a_val * b4.z;
        c4.w = a_val * b4.w;
        C_vec[col] = c4;
        col += stride;
    }
    
    // Handle remaining elements
    col = threadIdx.x + (vec_limit * 4);
    while (col < M) {
        C[row * M + col] = a_val * B[row * M + col];
        col += stride;
    }
}

// Forward function wraps the CUDA kernel
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.dim() == 1, "A must be a 1D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == B.size(0),
                "Dimension mismatch: A.size(0) must match B.size(0)");

    // Ensure inputs are contiguous
    A = A.contiguous();
    B = B.contiguous();

    int64_t N = A.size(0);
    int64_t M = B.size(1);

    // Create output tensor
    auto C = torch::empty({N, M}, B.options());

    // Launch kernel with one block per row
    const int threads = 32; // Reduced to one warp for better occupancy
    diag_matmul_warp_kernel<<<N, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N,
        M
    );

    return C;
}

// Pybind11 module definition
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized diagonal matrix multiplication using warp primitives");
}