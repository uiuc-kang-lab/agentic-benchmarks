#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

// Kernel to add bias to each output channel
__global__ void add_bias_kernel(
    float* output,
    const float* bias,
    int total,
    int C_out,
    int H_out,
    int W_out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= total) return;
    int oc = (index / (H_out * W_out)) % C_out;
    output[index] += bias[oc];
}

// Optimized kernel using shared memory for reduction
__global__ void conv_transpose2d_kernel_shared(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    float* __restrict__ output,
    int N,
    int C_in, int H_in, int W_in,
    int C_out,
    int K,
    int stride,
    int padding,
    int H_out, int W_out) {

    extern __shared__ float shared_mem[];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int bdim = blockDim.x;

    int n = bid / (C_out * H_out * W_out);
    int oc = (bid / (H_out * W_out)) % C_out;
    int h_out = (bid / W_out) % H_out;
    int w_out = bid % W_out;

    float sum = 0.0f;

    for (int c = 0; c < C_in; ++c) {
        for (int k_i = 0; k_i < K; ++k_i) {
            for (int k_j = 0; k_j < K; ++k_j) {
                int in_h = h_out + padding - k_i;
                int in_w = w_out + padding - k_j;
                if (in_h % stride != 0 || in_w % stride != 0) continue;
                in_h /= stride;
                in_w /= stride;
                if (in_h < 0 || in_h >= H_in || in_w < 0 || in_w >= W_in) continue;

                int input_idx = n * (C_in * H_in * W_in) + c * (H_in * W_in) + in_h * W_in + in_w;
                int weight_idx = c * (C_out * K * K) + oc * (K * K) + k_i * K + k_j;
                sum += input[input_idx] * weight[weight_idx];
            }
        }
    }

    // Store partial sum in shared memory
    shared_mem[tid] = sum;
    __syncthreads();

    // Perform reduction within the block
    for (int s = bdim / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_mem[tid] += shared_mem[tid + s];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        int out_idx = n * (C_out * H_out * W_out) + oc * (H_out * W_out) + h_out * W_out + w_out;
        output[out_idx] = shared_mem[0];
    }
}

// Forward function definition
torch::Tensor conv_transpose2d_forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int64_t stride,
    int64_t padding,
    int64_t output_padding,
    int64_t groups) {

    // Ensure inputs are on CUDA and contiguous
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA");
    TORCH_CHECK(x.is_contiguous(), "Input tensor must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "Weight tensor must be contiguous");

    if (bias.has_value()) {
        TORCH_CHECK(bias.value().is_cuda(), "Bias tensor must be on CUDA");
        TORCH_CHECK(bias.value().is_contiguous(), "Bias tensor must be contiguous");
    }

    // x: [N, C_in, H_in, W_in]
    auto x_sizes = x.sizes();
    int N = x_sizes[0];
    int C_in = x_sizes[1];
    int H_in = x_sizes[2];
    int W_in = x_sizes[3];

    // weight: [C_in, C_out, K, K]
    auto w_sizes = weight.sizes();
    int C_out = w_sizes[1];
    int K = w_sizes[2];

    // Compute output dimensions for transposed convolution
    int H_out = (H_in - 1) * stride - 2 * padding + K + output_padding;
    int W_out = (W_in - 1) * stride - 2 * padding + K + output_padding;

    auto output = torch::empty({N, C_out, H_out, W_out}, x.options());

    int total_output = N * C_out * H_out * W_out;
    int block_size = 256;
    int grid_size = total_output;
    size_t shared_mem_size = block_size * sizeof(float);

    // Launch the optimized kernel
    conv_transpose2d_kernel_shared<<<grid_size, block_size, shared_mem_size>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        output.data_ptr<float>(),
        N, C_in, H_in, W_in,
        C_out, K, stride, padding, H_out, W_out
    );
    hipDeviceSynchronize();

    // If bias is provided, add it using a separate kernel
    if (bias.has_value()) {
        int block_bias = 256;
        int grid_bias = (total_output + block_bias - 1) / block_bias;
        add_bias_kernel<<<grid_bias, block_bias>>>(
            output.data_ptr<float>(),
            bias.value().data_ptr<float>(),
            total_output, C_out, H_out, W_out
        );
        hipDeviceSynchronize();
    }

    return output;
}

// Pybind11 module definition
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward, "ConvTranspose2d forward with shared memory optimization (CUDA)");
}