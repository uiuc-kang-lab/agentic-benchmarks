#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define NUM_STREAMS 4

__global__ void avg_pool1d_kernel(
    const float *input,
    float *output,
    int kernel_size,
    int stride,
    int padding,
    int input_length,
    int output_length,
    int batch_size,
    int in_channels,
    int batch_offset,
    int batch_chunk_size) {

    int o = blockIdx.x * blockDim.x + threadIdx.x;
    int channel = blockIdx.y;
    int batch = blockIdx.z + batch_offset;

    if (o >= output_length || channel >= in_channels || batch >= (batch_offset + batch_chunk_size)) return;

    float sum = 0.0f;
    for (int k = 0; k < kernel_size; ++k) {
        int pos_padded = o * stride + k;
        int pos_input = pos_padded - padding;
        
        if (pos_input >= 0 && pos_input < input_length) {
            int input_idx = batch * in_channels * input_length + channel * input_length + pos_input;
            sum += __ldg(&input[input_idx]);
        }
    }

    output[batch * in_channels * output_length + channel * output_length + o] = sum / kernel_size;
}

torch::Tensor avg_pool1d_forward(
    const torch::Tensor &x,
    int kernel_size,
    int stride,
    int padding) {
    
    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(kernel_size > 0 && stride > 0 && padding >= 0, "Invalid kernel parameters");

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int input_length = x.size(2);
    int output_length = (input_length + 2 * padding - kernel_size) / stride + 1;

    auto output = torch::empty({batch_size, in_channels, output_length}, x.options());

    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    dim3 threads(256);
    int batch_chunk_size = (batch_size + NUM_STREAMS - 1) / NUM_STREAMS;

    for (int i = 0; i < NUM_STREAMS; i++) {
        int batch_offset = i * batch_chunk_size;
        int current_chunk_size = min(batch_chunk_size, batch_size - batch_offset);
        
        if (current_chunk_size <= 0) break;

        dim3 grid(
            (output_length + threads.x - 1) / threads.x,
            in_channels,
            current_chunk_size
        );

        avg_pool1d_kernel<<<grid, threads, 0, streams[i]>>>(
            x.data_ptr<float>(),
            output.data_ptr<float>(),
            kernel_size,
            stride,
            padding,
            input_length,
            output_length,
            batch_size,
            in_channels,
            batch_offset,
            current_chunk_size
        );
    }

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &avg_pool1d_forward, "1D Average Pooling forward (CUDA)");
}