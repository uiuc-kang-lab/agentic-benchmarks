#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_softplus(scalar_t x) {
    if (x > 20.0) {
        return x;
    } else if (x < -20.0) {
        return exp(x);
    } else {
        return log1p(exp(x));
    }
}

template <typename scalar_t>
__global__ void softplus_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    // Grid-stride loop pattern
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    // Each thread processes multiple elements in a strided fashion
    for (int idx = tid; idx < size; idx += stride) {
        const scalar_t x = input[idx];
        output[idx] = compute_softplus(x);
    }
}

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    
    // Use smaller thread blocks for better occupancy
    const int threads = 128;
    
    // Calculate optimal number of blocks based on SM count
    int multiProcessorCount;
    hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount, 0);
    // Aim for 4 blocks per SM for better distribution
    const int blocks = min(4 * multiProcessorCount, (size + threads - 1) / threads);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "softplus_forward_cuda", ([&] {
        softplus_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}