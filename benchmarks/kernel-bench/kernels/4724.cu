#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>

// CUDA kernel for computing sum of squares with efficient thread and block mapping
__global__ void compute_norm_kernel_efficient(const float* input, float* norm_out, int numel) {
    __shared__ float sdata[256];
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;
    float sum = 0.0f;

    // Each thread computes its partial sum
    for (int i = idx; i < numel; i += blockDim.x * gridDim.x) {
        float val = input[i];
        sum += val * val;
    }

    // Write partial sum to shared memory
    sdata[tid] = sum;
    __syncthreads();

    // First reduction stage: combine 256 -> 128
    if (tid < 128) {
        sdata[tid] += sdata[tid + 128];
    }
    __syncthreads();

    // Second stage: reduce 128 -> 64 using shared memory;
    if (tid < 64) {
        volatile float* vsdata = sdata;
        vsdata[tid] += vsdata[tid + 64];
        float val = vsdata[tid];
        for (int offset = 32; offset > 0; offset /= 2) {
            val += __shfl_down_sync(0xffffffff, val, offset);
        }
        if (tid == 0) {
            atomicAdd(norm_out, val);
        }
    }
}

// CUDA kernel for tensor normalization
__global__ void normalize_kernel(const float* input, float* output, float norm, int numel) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numel) {
        output[idx] = input[idx] / norm;
    }
}

// Host function interfacing with PyTorch
torch::Tensor forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be on GPU");
    TORCH_CHECK(input.is_contiguous(), "Input tensor must be contiguous");
    TORCH_CHECK(input.scalar_type() == torch::kFloat32, "Input must be float32");

    auto output = torch::empty_like(input);
    auto norm_tensor = torch::zeros({1}, input.options());

    const float* input_ptr = input.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();
    float* norm_ptr = norm_tensor.data_ptr<float>();
    int numel = input.numel();

    const int threads = 256;
    const int blocks = min(65535, (numel + threads - 1) / threads);

    // Launch kernel to compute the sum of squares with efficient mapping
    compute_norm_kernel_efficient<<<blocks, threads>>>(input_ptr, norm_ptr, numel);
    
    // Retrieve the computed sum and calculate the Frobenius norm
    float norm_val;
    hipMemcpy(&norm_val, norm_ptr, sizeof(float), hipMemcpyDeviceToHost);
    norm_val = sqrt(norm_val);

    // Launch kernel to normalize the tensor
    normalize_kernel<<<blocks, threads>>>(input_ptr, output_ptr, norm_val, numel);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Frobenius norm normalization");
}
