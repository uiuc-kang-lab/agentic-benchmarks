#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device function to compute softplus in a numerically stable way
template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_softplus(scalar_t x) {
    if (x > 20.0) {
        return x;
    } else if (x < -20.0) {
        return exp(x);
    } else {
        const scalar_t exp_x = exp(x);
        return log1p(exp_x);
    }
}

// Kernel using grid-stride loop to distribute workload evenly
template <typename scalar_t>
__global__ void softplus_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = idx; i < size; i += stride) {
        const scalar_t x = input[i];
        output[i] = compute_softplus(x);
    }
}

// CUDA forward function
torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    const int threads = 256;
    const int blocks = (size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "softplus_forward_cuda", ([&] {
        softplus_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}
