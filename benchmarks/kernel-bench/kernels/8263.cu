#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/util/Optional.h>
#include <vector>

namespace py = pybind11;

// Combined CUDA kernel with improved thread mapping and stream support
__global__ void optimized_conv1d_kernel(
    const float* __restrict__ x,
    const float* __restrict__ w,
    const float* __restrict__ bias_ptr, // may be null
    float* __restrict__ y,
    int start_n,   // starting batch index for this chunk
    int N_chunk,   // number of samples in this chunk
    int C_in,
    int L_in,
    int C_out,
    int K,
    int stride,
    int padding,
    int dilation,
    int groups,
    int L_out
) {
    // Improved thread mapping: each block handles an output channel and a tile of output positions
    int out_ch = blockIdx.x;                     // output channel index
    int out_pos = blockIdx.y * blockDim.x + threadIdx.x; // output position within L_out
    int n_local = blockIdx.z;                      // local batch index within this chunk

    if (out_pos >= L_out || n_local >= N_chunk)
        return;

    int n = start_n + n_local; // global batch index

    // Determine channel grouping
    int group_size_out = C_out / groups;
    int group_size_in  = C_in  / groups;
    int group_idx = out_ch / group_size_out;

    float result = 0.0f;

    // Compute convolution for this output element
    for (int local_in_ch = 0; local_in_ch < group_size_in; local_in_ch++) {
        int in_ch = group_idx * group_size_in + local_in_ch;
        for (int k = 0; k < K; k++) {
            int in_pos = out_pos * stride + k * dilation - padding;
            if (in_pos >= 0 && in_pos < L_in) {
                float x_val = x[n * (C_in * L_in) + in_ch * L_in + in_pos];
                float w_val = w[out_ch * (group_size_in * K) + local_in_ch * K + k];
                result += x_val * w_val;
            }
        }
    }

    if (bias_ptr) {
        result += bias_ptr[out_ch];
    }

    y[n * (C_out * L_out) + out_ch * L_out + out_pos] = result;
}

// Host implementation that uses streams and improved kernel
at::Tensor optimized_conv1d_forward(
    const at::Tensor& x,
    const at::Tensor& weight,
    c10::optional<at::Tensor> bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups
) {
    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.scalar_type() == at::kFloat, "x must be float32");
    TORCH_CHECK(weight.scalar_type() == at::kFloat, "weight must be float32");

    auto x_sizes = x.sizes();
    int64_t N_total = x_sizes[0];
    int64_t C_in = x_sizes[1];
    int64_t L_in = x_sizes[2];

    auto w_sizes = weight.sizes();
    int64_t C_out = w_sizes[0];
    int64_t K = w_sizes[2];

    int64_t L_out = (L_in + 2 * padding - dilation * (K - 1) - 1) / stride + 1;
    TORCH_CHECK(L_out > 0, "Calculated output length is non-positive.");

    auto y = torch::empty({N_total, C_out, L_out}, x.options().dtype(at::kFloat));

    const float* bias_ptr = nullptr;
    if (bias_opt.has_value() && bias_opt.value().defined()) {
        TORCH_CHECK(bias_opt.value().is_cuda(), "bias must be a CUDA tensor if provided");
        TORCH_CHECK(bias_opt.value().scalar_type() == at::kFloat, "bias must be float32");
        bias_ptr = bias_opt.value().data_ptr<float>();
    }

    int num_streams = (N_total < 4) ? N_total : 4; // use up to 4 streams
    int chunk_size = (N_total + num_streams - 1) / num_streams;  // ceiling division

    std::vector<hipStream_t> streams(num_streams);
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
    }

    dim3 blockSize(256);
    dim3 gridSize;
    gridSize.x = C_out;
    gridSize.y = (L_out + blockSize.x - 1) / blockSize.x;

    for (int i = 0; i < num_streams; i++) {
        int start_n = i * chunk_size;
        if (start_n >= N_total) break;
        int current_chunk = std::min(chunk_size, (int)(N_total - start_n));

        dim3 grid = gridSize;
        grid.z = current_chunk;

        optimized_conv1d_kernel<<<grid, blockSize, 0, streams[i]>>>(
            x.data_ptr<float>(),
            weight.data_ptr<float>(),
            bias_ptr,
            y.data_ptr<float>(),
            start_n,
            current_chunk,
            (int)C_in,
            (int)L_in,
            (int)C_out,
            (int)K,
            (int)stride,
            (int)padding,
            (int)dilation,
            (int)groups,
            (int)L_out
        );

        hipError_t err = hipGetLastError();
        TORCH_CHECK(err == hipSuccess, "optimized_conv1d_kernel launch failed: ", hipGetErrorString(err));
    }

    for (int i = 0; i < num_streams; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        [](at::Tensor x,
           at::Tensor weight,
           py::object bias_obj,
           int64_t stride,
           int64_t padding,
           int64_t dilation,
           int64_t groups) {
            c10::optional<at::Tensor> bias;
            if (!bias_obj.is_none()) {
                bias = bias_obj.cast<at::Tensor>();
            }
            return optimized_conv1d_forward(x, weight, bias, stride, padding, dilation, groups);
        },
        "Optimized 1D Convolution forward (CUDA) with improved thread mapping and stream pipelining"
    );
}