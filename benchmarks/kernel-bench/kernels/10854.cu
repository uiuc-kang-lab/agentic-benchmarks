#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Inline device function that computes the masked cumulative sum for one row using
// a branchless update. This merges modularity with efficiency.
template <typename scalar_t>
__device__ __forceinline__ void compute_masked_cumsum(const scalar_t* __restrict__ x_row,
                                                         const bool* __restrict__ mask_row,
                                                         scalar_t* __restrict__ output_row,
                                                         int64_t L) {
    scalar_t sum = static_cast<scalar_t>(0);
    for (int64_t i = 0; i < L; ++i) {
        // Branchless update avoids divergence: mask_row[i] is implicitly casted to 0 or 1
        sum += x_row[i] * static_cast<scalar_t>(mask_row[i]);
        output_row[i] = sum;
    }
}


// CUDA kernel that processes one row per thread, delegating the work
// to the inlined compute_masked_cumsum function
template <typename scalar_t>
__global__ void merged_modular_masked_cumsum_kernel(
    const scalar_t* __restrict__ x,
    const bool* __restrict__ mask,
    scalar_t* __restrict__ output,
    int64_t N,
    int64_t L) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        compute_masked_cumsum<scalar_t>(x + idx * L,
                                         mask + idx * L,
                                         output + idx * L,
                                         L);
    }
}

// Macros for error checking
#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Host function that prepares input, launches the CUDA kernel, and permutes the output back
torch::Tensor merged_modular_masked_cumsum(const torch::Tensor& x,
                                             const torch::Tensor& mask,
                                             int64_t dim) {
    CHECK_INPUT(x);
    CHECK_INPUT(mask);
    TORCH_CHECK(x.sizes() == mask.sizes(), "x and mask must have the same shape");
    TORCH_CHECK(mask.scalar_type() == torch::kBool, "mask must be a boolean tensor");

    // Adjust negative dimension
    if (dim < 0) {
        dim += x.dim();
    }
    TORCH_CHECK(dim >= 0 && dim < x.dim(), "Invalid dimension");

    // Permute dimensions to bring the target dimension to the last position
    std::vector<int64_t> perm;
    for (int64_t i = 0; i < x.dim(); ++i) {
        if (i != dim) {
            perm.push_back(i);
        }
    }
    perm.push_back(dim);

    auto x_permuted = x.permute(perm).contiguous();
    auto mask_permuted = mask.permute(perm).contiguous();

    // Reshape into 2D: (N rows x L columns)
    int64_t L = x_permuted.size(-1);
    int64_t N = x_permuted.numel() / L;

    auto x_flat = x_permuted.view({N, L});
    auto mask_flat = mask_permuted.view({N, L});
    auto output_flat = torch::empty_like(x_flat);

    // Launch CUDA kernel with one thread per row
    const int threads = 256;
    const int blocks = (N + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "merged_modular_masked_cumsum_cuda", ([&] {
        merged_modular_masked_cumsum_kernel<scalar_t><<<blocks, threads>>>(
            x_flat.data_ptr<scalar_t>(),
            mask_flat.data_ptr<bool>(),
            output_flat.data_ptr<scalar_t>(),
            N,
            L
        );
    }));

    // Reshape and inverse-permute back to the original dimension order
    auto output_permuted = output_flat.view(x_permuted.sizes());
    std::vector<int64_t> inv_perm(perm.size());
    for (size_t i = 0; i < perm.size(); ++i) {
        inv_perm[perm[i]] = i;
    }
    auto output = output_permuted.permute(inv_perm);
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &merged_modular_masked_cumsum, "Merged Modular Masked Cumulative Sum (CUDA)");
}
