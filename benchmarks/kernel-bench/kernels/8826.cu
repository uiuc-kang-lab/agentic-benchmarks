#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device function to compute a single output element
__device__ float compute_output_element(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    int N, int C_in, int C_out, int L_in, int L_out, int K_w,
    int stride, int padding, int dilation,
    int n, int c_out, int l_out)
{
    float value = (bias != nullptr) ? bias[c_out] : 0.0f;

    for (int c_in = 0; c_in < C_in; ++c_in) {
        for (int k_w = 0; k_w < K_w; ++k_w) {
            int l_in_nom = l_out + padding - k_w * dilation;
            int l_in = l_in_nom / stride;
            // Use a single condition to minimize warp divergence
            if (l_in_nom % stride == 0 && l_in >= 0 && l_in < L_in) {
                float x_val = x[n * C_in * L_in + c_in * L_in + l_in];
                float w_val = weight[c_in * C_out * K_w + c_out * K_w + k_w];
                value += x_val * w_val;
            }
        }
    }
    return value;
}

// CUDA kernel for ConvTranspose1D
__global__ void conv_transpose1d_kernel(
    const float* __restrict__ x,       // [N, C_in, L_in]
    const float* __restrict__ weight,  // [C_in, C_out, K_w]
    const float* __restrict__ bias,    // [C_out] or nullptr
    float* __restrict__ y,             // [N, C_out, L_out]
    int N, int C_in, int C_out, int L_in, int L_out, int K_w,
    int stride, int padding, int dilation)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = N * C_out * L_out;
    if (index >= total_elements) return;

    int l_out = index % L_out;
    int c_out = (index / L_out) % C_out;
    int n = index / (L_out * C_out);

    y[n * C_out * L_out + c_out * L_out + l_out] = compute_output_element(
        x, weight, bias, N, C_in, C_out, L_in, L_out, K_w, stride, padding, dilation, n, c_out, l_out);
}

torch::Tensor conv_transpose1d_forward(
    py::object x_obj,            // x: torch.Tensor
    py::object weight_obj,       // weight: torch.Tensor or torch.nn.Parameter
    py::object bias_obj = py::none(),  // bias: torch.Tensor or None
    int64_t stride = 1,
    int64_t padding = 0,
    int64_t dilation = 1)
{
    // Convert py::object to torch::Tensor
    torch::Tensor x = x_obj.cast<torch::Tensor>();
    torch::Tensor weight = weight_obj.cast<torch::Tensor>();

    // Ensure inputs are contiguous and on CUDA device
    x = x.contiguous();
    weight = weight.contiguous();

    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA device");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA device");

    // Handle bias
    float* bias_ptr = nullptr;
    if (!bias_obj.is_none()) {
        torch::Tensor bias = bias_obj.cast<torch::Tensor>();
        bias = bias.contiguous();
        TORCH_CHECK(bias.is_cuda(), "Bias tensor must be on CUDA device");
        bias_ptr = bias.data_ptr<float>();
    }

    // Get input dimensions
    int N = x.size(0);
    int C_in = x.size(1);
    int L_in = x.size(2);
    int K_w = weight.size(2);
    int C_out = weight.size(1);

    // Compute output length
    int L_out = (L_in - 1) * stride - 2 * padding + dilation * (K_w - 1) + 1;

    // Allocate output tensor
    auto y = torch::empty({N, C_out, L_out}, x.options());

    // Launch kernel with optimized block size
    int total_elements = N * C_out * L_out;
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;

    conv_transpose1d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        y.data_ptr<float>(),
        N, C_in, C_out, L_in, L_out, K_w,
        stride, padding, dilation);

    // Check for CUDA errors
    TORCH_CHECK(hipGetLastError() == hipSuccess, "CUDA kernel failed");

    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        &conv_transpose1d_forward,
        "Conv Transpose1D forward (CUDA)",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = 1,
        py::arg("padding") = 0,
        py::arg("dilation") = 1);
}
