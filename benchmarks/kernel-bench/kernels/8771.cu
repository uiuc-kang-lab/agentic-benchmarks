#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Constant memory for weights and bias
__constant__ float c_weight[16384];  // Adjust size based on expected maximum weight size
__constant__ float c_bias[1024];     // Adjust size based on expected maximum bias size

__global__ void conv_transpose1d_kernel_const_mem(
    const float* __restrict__ x,
    float* __restrict__ y,
    int N, int C_in, int C_out, int L_in, int L_out, int K_w,
    int stride, int padding, int dilation,
    bool has_bias) {

    int total_elements = N * C_out * L_out;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride_elements = blockDim.x * gridDim.x;

    for (int idx = index; idx < total_elements; idx += stride_elements) {
        int l_out = idx % L_out;
        int c_out = (idx / L_out) % C_out;
        int n = idx / (L_out * C_out);

        float value = has_bias ? c_bias[c_out] : 0.0f;

        for (int c_in = 0; c_in < C_in; ++c_in) {
            int x_base = n * C_in * L_in + c_in * L_in;
            int w_base = c_in * C_out * K_w + c_out * K_w;

            #pragma unroll 4
            for (int k_w = 0; k_w < K_w; ++k_w) {
                int l_in_nom = l_out + padding - k_w * dilation;
                int l_in = l_in_nom / stride;
                
                if ((l_in_nom % stride == 0) && (l_in >= 0) && (l_in < L_in)) {
                    float x_val = x[x_base + l_in];
                    float w_val = c_weight[w_base + k_w];
                    value += x_val * w_val;
                }
            }
        }
        y[n * C_out * L_out + c_out * L_out + l_out] = value;
    }
}

torch::Tensor conv_transpose1d_forward(
    py::object x_obj,
    py::object weight_obj,
    py::object bias_obj = py::none(),
    int64_t stride = 1,
    int64_t padding = 0,
    int64_t dilation = 1) {

    torch::Tensor x = x_obj.cast<torch::Tensor>().contiguous();
    torch::Tensor weight = weight_obj.cast<torch::Tensor>().contiguous();

    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA device");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA device");

    bool has_bias = !bias_obj.is_none();
    
    // Copy weight to constant memory
    int weight_size = weight.numel() * sizeof(float);
    TORCH_CHECK(weight_size <= 16384 * sizeof(float), "Weight tensor too large for constant memory");
    hipMemcpyToSymbol(HIP_SYMBOL(c_weight), weight.data_ptr<float>(), weight_size);

    // Copy bias to constant memory if present
    if (has_bias) {
        torch::Tensor bias = bias_obj.cast<torch::Tensor>().contiguous();
        TORCH_CHECK(bias.is_cuda(), "Bias tensor must be on CUDA device");
        hipMemcpyToSymbol(HIP_SYMBOL(c_bias), bias.data_ptr<float>(), bias.numel() * sizeof(float));
    }

    int N = x.size(0);
    int C_in = x.size(1);
    int L_in = x.size(2);
    int K_w = weight.size(2);
    int C_out = weight.size(1);
    int L_out = (L_in - 1) * stride - 2 * padding + dilation * (K_w - 1) + 1;

    auto y = torch::empty({N, C_out, L_out}, x.options());

    int threads = 256;
    int blocks = (N * C_out * L_out + threads - 1) / threads;

    conv_transpose1d_kernel_const_mem<<<blocks, threads>>>(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        N, C_in, C_out, L_in, L_out, K_w,
        stride, padding, dilation,
        has_bias);

    TORCH_CHECK(hipGetLastError() == hipSuccess, "CUDA kernel failed");
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        &conv_transpose1d_forward,
        "Constant Memory Conv Transpose1D forward (CUDA)",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = 1,
        py::arg("padding") = 0,
        py::arg("dilation") = 1);
}