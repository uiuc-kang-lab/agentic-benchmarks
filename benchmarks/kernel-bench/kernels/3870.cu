#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Kernel operates on a chunk of the tensor. The offset allows indexing into the full input.
__global__ void softsign_kernel(const float* x, float* out, int offset, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        int global_idx = offset + idx;
        out[global_idx] = x[global_idx] / (1.0f + fabsf(x[global_idx]));
    }
}

// The forward function splits work into chunks and launches each chunk asynchronously on its own CUDA stream.
// This enables overlapping of kernel execution with asynchronous memory operations (if any),
// thereby pipelining the workload and reducing the overall runtime on advanced GPUs such as the NVIDIA H100.

torch::Tensor forward(torch::Tensor x) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int total_elements = x.numel();

    // Set chunk size (number of elements per stream). This size can be tuned for best performance.
    int chunk_size = 1 << 20; // Example: 1M elements per chunk
    int num_chunks = (total_elements + chunk_size - 1) / chunk_size;
    int threads = 1024;

    // Create a vector to hold CUDA streams
    std::vector<hipStream_t> streams(num_chunks);

    // Launch kernel for each chunk on its own stream
    for (int i = 0; i < num_chunks; i++) {
        hipStreamCreate(&streams[i]);
        int offset = i * chunk_size;
        int current_chunk = std::min(chunk_size, total_elements - offset);
        int blocks = (current_chunk + threads - 1) / threads;
        softsign_kernel<<<blocks, threads, 0, streams[i]>>>(x.data_ptr<float>(), out.data_ptr<float>(), offset, current_chunk);
    }

    // Ensure all streams have completed execution
    for (int i = 0; i < num_chunks; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Softsign activation (CUDA with stream overlap)");
}
