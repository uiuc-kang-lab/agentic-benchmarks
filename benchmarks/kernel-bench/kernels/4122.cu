#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

// Optimized kernel combining both approaches
template <typename scalar_t>
__global__ void optimized_hardtanh_kernel(const scalar_t* __restrict__ x,
                                          scalar_t* __restrict__ out,
                                          int64_t numel,
                                          scalar_t min_val,
                                          scalar_t max_val) {
  int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t stride = blockDim.x * gridDim.x;
  for (int64_t idx = i; idx < numel; idx += stride) {
    scalar_t val = x[idx];
    // Clamp between min_val and max_val using ternary operator for efficiency
    val = val < min_val ? min_val : (val > max_val ? max_val : val);
    out[idx] = val;
  }
}

at::Tensor forward_cuda(const at::Tensor& x, float min_val, float max_val) {
  auto out = at::empty_like(x);
  int64_t numel = x.numel();

  const int threads = 256;  // Tuned block size based on experiments
  const int blocks = (numel + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "hardtanh_cuda", ([&] {
    optimized_hardtanh_kernel<scalar_t><<<blocks, threads>>>(
        x.data_ptr<scalar_t>(),
        out.data_ptr<scalar_t>(),
        numel,
        static_cast<scalar_t>(min_val),
        static_cast<scalar_t>(max_val)
    );
  }));

  return out;
}

at::Tensor forward(const at::Tensor& x, float min_val, float max_val) {
  if (!x.is_cuda()) {
    throw std::invalid_argument("Input tensor must be a CUDA tensor");
  }
  return forward_cuda(x, min_val, max_val);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "HardTanh activation (CUDA)");
}