#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

namespace py = pybind11;

// This kernel implements conv_transpose2d using a gather approach. Each thread computes one output pixel
// by gathering contributions from all input channels and the kernel window. By doing so, no atomic
// operations are needed, reducing global memory contention.

__global__ void conv_transpose2d_forward_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int N,
    int in_channels,
    int in_h,
    int in_w,
    int out_channels,
    int kernel_h,
    int kernel_w,
    int out_h,
    int out_w,
    int stride_h,
    int stride_w,
    int pad_h,
    int pad_w,
    bool has_bias
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = N * out_channels * out_h * out_w;
    if (index >= total) return;

    // Decode the output index into (n, oc, out_y, out_x) assuming row-major layout
    int out_x = index % out_w;
    int tmp = index / out_w;
    int out_y = tmp % out_h;
    tmp = tmp / out_h;
    int oc = tmp % out_channels;
    int n = tmp / out_channels;

    float sum = has_bias ? bias[oc] : 0.0f;

    // For each input channel and kernel element, accumulate the contribution if the mapping is valid
    for (int ic = 0; ic < in_channels; ic++) {
        for (int ky = 0; ky < kernel_h; ky++) {
            for (int kx = 0; kx < kernel_w; kx++) {
                int t_y = out_y + pad_h - ky;
                int t_x = out_x + pad_w - kx;
                // Check alignment with stride
                if ((t_y % stride_h == 0) && (t_x % stride_w == 0)) {
                    int in_y = t_y / stride_h;
                    int in_x = t_x / stride_w;
                    if (in_y >= 0 && in_y < in_h && in_x >= 0 && in_x < in_w) {
                        int input_idx = ((n * in_channels + ic) * in_h + in_y) * in_w + in_x;
                        int weight_idx = ((ic * out_channels + oc) * kernel_h + ky) * kernel_w + kx;
                        sum += input[input_idx] * weight[weight_idx];
                    }
                }
            }
        }
    }

    output[index] = sum;
}


torch::Tensor conv_transpose2d_forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding
) {
    // Input shape: [N, in_channels, in_h, in_w]
    auto N = input.size(0);
    auto in_channels = input.size(1);
    auto in_h = input.size(2);
    auto in_w = input.size(3);

    // Weight shape: [in_channels, out_channels, kernel_h, kernel_w]
    auto out_channels = weight.size(1);
    auto kernel_h = weight.size(2);
    auto kernel_w = weight.size(3);

    int stride_h = stride[0];
    int stride_w = stride[1];
    int pad_h = padding[0];
    int pad_w = padding[1];

    // Compute output dimensions based on standard transposed convolution formula
    int out_h = (in_h - 1) * stride_h - 2 * pad_h + kernel_h;
    int out_w = (in_w - 1) * stride_w - 2 * pad_w + kernel_w;

    auto output = torch::zeros({N, out_channels, out_h, out_w}, input.options());

    int total = N * out_channels * out_h * out_w;
    int threads = 256;
    int blocks = (total + threads - 1) / threads;

    bool has_bias = (bias.has_value() && bias.value().numel() > 0);
    const float* bias_ptr = has_bias ? bias.value().data_ptr<float>() : nullptr;

    conv_transpose2d_forward_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        N,
        in_channels,
        in_h,
        in_w,
        out_channels,
        kernel_h,
        kernel_w,
        out_h,
        out_w,
        stride_h,
        stride_w,
        pad_h,
        pad_w,
        has_bias
    );

    return output;
}


// Entry point from Python
torch::Tensor conv_transpose2d_forward(
    torch::Tensor input,
    torch::Tensor weight,
    py::object bias_obj,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding
) {
    c10::optional<torch::Tensor> bias = c10::nullopt;
    if (!bias_obj.is_none()) {
        bias = bias_obj.cast<torch::Tensor>();
    }
    return conv_transpose2d_forward_cuda(input, weight, bias, stride, padding);
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward, "Conv Transpose 2D forward (gather approach)",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride"),
          py::arg("padding"));
}
