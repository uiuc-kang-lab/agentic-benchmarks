#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template<typename scalar_t, int KERNEL_SIZE>
__device__ __forceinline__ scalar_t compute_max_aligned(
    const scalar_t* __restrict__ input,
    const int base_offset,
    const int ih_start,
    const int iw_start,
    const int input_height,
    const int input_width,
    const int input_stride_h,
    const int stride,
    const int dilation
) {
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    
    #pragma unroll
    for (int kh = 0; kh < KERNEL_SIZE; kh++) {
        const int ih = ih_start + kh * dilation;
        if (ih >= 0 && ih < input_height) {
            const int h_offset = ih * input_stride_h;
            
            // Ensure aligned access for float4 loads when possible
            const int aligned_iw = (iw_start + 3) & ~3;
            const int end_aligned_iw = iw_start + KERNEL_SIZE * dilation;
            
            // Handle pre-aligned elements
            #pragma unroll
            for (int iw = iw_start; iw < min(aligned_iw, end_aligned_iw); iw++) {
                if (iw >= 0 && iw < input_width) {
                    max_val = max(max_val, __ldg(&input[base_offset + h_offset + iw]));
                }
            }
            
            // Vector loads for aligned portion
            if (sizeof(scalar_t) == 4 && aligned_iw + 4 <= end_aligned_iw) {
                const float4* aligned_input = reinterpret_cast<const float4*>(&input[base_offset + h_offset + aligned_iw]);
                float4 vec_val = __ldg(aligned_input);
                max_val = max(max_val, max(max(vec_val.x, vec_val.y), max(vec_val.z, vec_val.w)));
            }
            
            // Handle remaining elements
            #pragma unroll
            for (int iw = aligned_iw + 4; iw < end_aligned_iw; iw++) {
                if (iw >= 0 && iw < input_width) {
                    max_val = max(max_val, __ldg(&input[base_offset + h_offset + iw]));
                }
            }
        }
    }
    return max_val;
}

template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int stride_x = blockDim.x * gridDim.x;
    
    const int total_elements = batch_size * channels * output_height * output_width;
    
    for (int index = tid; index < total_elements; index += stride_x) {
        const int ow = index % output_width;
        const int oh = (index / output_width) % output_height;
        const int c = (index / (output_width * output_height)) % channels;
        const int b = index / (output_width * output_height * channels);

        const int input_stride_batch = channels * input_height * input_width;
        const int input_stride_channel = input_height * input_width;
        const int input_stride_h = input_width;

        const int base_offset = b * input_stride_batch + c * input_stride_channel;
        const int ih_start = oh * stride - padding;
        const int iw_start = ow * stride - padding;

        scalar_t max_val;
        
        switch(kernel_size) {
            case 2:
                max_val = compute_max_aligned<scalar_t, 2>(
                    input, base_offset, ih_start, iw_start,
                    input_height, input_width, input_stride_h,
                    stride, dilation);
                break;
            case 3:
                max_val = compute_max_aligned<scalar_t, 3>(
                    input, base_offset, ih_start, iw_start,
                    input_height, input_width, input_stride_h,
                    stride, dilation);
                break;
            default:
                max_val = compute_max_aligned<scalar_t, 4>(
                    input, base_offset, ih_start, iw_start,
                    input_height, input_width, input_stride_h,
                    stride, dilation);
        }
        
        output[index] = max_val;
    }
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int threads = 256;
    const int blocks = min(65535, (batch_size * channels * output_height * output_width + threads - 1) / threads);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}