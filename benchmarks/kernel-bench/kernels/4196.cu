#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

template <typename scalar_t>
__global__ void hardtanh_grid_stride_kernel(const scalar_t* __restrict__ x,
                                          scalar_t* __restrict__ out,
                                          int64_t numel,
                                          scalar_t min_val,
                                          scalar_t max_val) {
  const int64_t grid_stride = blockDim.x * gridDim.x;
  int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  
  while (i < numel) {
    scalar_t val = x[i];
    out[i] = max(min(val, max_val), min_val);
    i += grid_stride;
  }
}

at::Tensor forward_cuda(const at::Tensor& x, float min_val, float max_val) {
  auto out = at::empty_like(x);
  int64_t numel = x.numel();

  const int threads = 256;
  int blocks = (numel + threads - 1) / threads;
  auto* device_properties = at::cuda::getCurrentDeviceProperties();
  blocks = std::min(blocks, device_properties->maxGridSize[0]);

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "hardtanh_grid_stride_cuda", ([&] {
    hardtanh_grid_stride_kernel<scalar_t><<<blocks, threads>>>(
        x.data_ptr<scalar_t>(),
        out.data_ptr<scalar_t>(),
        numel,
        static_cast<scalar_t>(min_val),
        static_cast<scalar_t>(max_val)
    );
  }));

  return out;
}

at::Tensor forward(const at::Tensor& x, float min_val, float max_val) {
  if (!x.is_cuda()) {
    throw std::invalid_argument("Input tensor must be a CUDA tensor");
  }
  return forward_cuda(x, min_val, max_val);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "HardTanh activation (CUDA)");
}