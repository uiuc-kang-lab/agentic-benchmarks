#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// This CUDA kernel performs 2D convolution on asymmetric input with a square kernel.
// It uses #pragma unroll on the nested loops that iterate over the kernel dimensions to
// reduce loop overhead, which is especially effective when kernel_size is small.

__global__ void conv2d_forward_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int out_height,
    int out_width,
    int kernel_size,
    int stride,
    int padding,
    int dilation) {

    // Identify which output element this thread is responsible for.
    int n = blockIdx.x;       // batch index
    int oc = blockIdx.y;      // output channel index
    int pixel_idx = blockIdx.z * blockDim.x + threadIdx.x;
    if (pixel_idx >= out_height * out_width) return;

    int out_y = pixel_idx / out_width;
    int out_x = pixel_idx % out_width;

    float sum = 0.0f;

    // Loop over input channels
    for (int ic = 0; ic < in_channels; ++ic) {
        // Manually unroll kernel height and width loops
        #pragma unroll
        for (int ky = 0; ky < kernel_size; ++ky) {
            #pragma unroll
            for (int kx = 0; kx < kernel_size; ++kx) {
                int in_y = out_y * stride - padding + ky * dilation;
                int in_x = out_x * stride - padding + kx * dilation;

                // Boundary check
                if (in_y >= 0 && in_y < in_height && in_x >= 0 && in_x < in_width) {
                    int input_idx = n * in_channels * in_height * in_width
                                  + ic * in_height * in_width
                                  + in_y * in_width
                                  + in_x;
                    int weight_idx = oc * in_channels * kernel_size * kernel_size
                                   + ic * kernel_size * kernel_size
                                   + ky * kernel_size
                                   + kx;
                    sum += input[input_idx] * weight[weight_idx];
                }
            }
        }
    }

    // Add bias if provided
    if (bias) {
        sum += bias[oc];
    }
    
    int output_idx = n * out_channels * out_height * out_width
                   + oc * out_height * out_width
                   + out_y * out_width
                   + out_x;
    output[output_idx] = sum;
}


// Host function that prepares the tensors and launches the CUDA kernel
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    // Input checks
    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    // This implementation supports groups==1 only
    TORCH_CHECK(groups == 1, "Only groups==1 is supported in this optimized kernel");

    // Extract dimensions
    const int batch = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);

    const int out_channels = weight.size(0);
    const int kernel_size = weight.size(2); // square kernel assumed (weight.size(2) == weight.size(3))

    // Compute output dimensions
    const int out_height = (in_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;
    const int out_width  = (in_width  + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;

    auto output = torch::zeros({batch, out_channels, out_height, out_width}, x.options());

    // Determine kernel launch parameters
    const int output_pixels = out_height * out_width;
    const int threads = 256;
    const int blocks_z = (output_pixels + threads - 1) / threads;
    dim3 grid(batch, out_channels, blocks_z);
    dim3 block(threads);

    const float* input_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = (bias.has_value()) ? bias.value().data_ptr<float>() : nullptr;
    float* output_ptr = output.data_ptr<float>();

    // Launch the CUDA convolution kernel
    conv2d_forward_kernel<<<grid, block>>>(
        input_ptr, weight_ptr, bias_ptr, output_ptr,
        batch, in_channels, out_channels,
        in_height, in_width, out_height, out_width,
        kernel_size, stride, padding, dilation);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized CUDA forward function for 2D convolution with manual loop unrolling");
}
