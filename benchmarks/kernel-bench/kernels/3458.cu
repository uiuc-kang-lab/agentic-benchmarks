#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <algorithm>

// Device function: GELU activation applied elementwise
// Specializations for float and double

template <typename scalar_t>
__device__ inline scalar_t gelu_function(scalar_t x);

template <>
__device__ inline float gelu_function<float>(float x) {
    // GELU(x) = x * 0.5 * (1 + erf(x / sqrt(2)))
    return x * 0.5f * (1.0f + erff(x / 1.4142135623730951f));
}

template <>
__device__ inline double gelu_function<double>(double x) {
    return x * 0.5 * (1.0 + erf(x / 1.4142135623730951));
}

// Kernel to compute GELU over a chunk using a grid-stride loop
template <typename scalar_t>
__global__ void gelu_kernel_compute(const scalar_t* __restrict__ in,
                                      scalar_t* __restrict__ out,
                                      size_t numel) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = idx; i < numel; i += stride) {
        scalar_t val = in[i];
        out[i] = gelu_function<scalar_t>(val);
    }
}

// Forward function with pipelined overlapping of memory transfers and computation
// Uses two separate streams (one for memcpy and one for compute) with double buffering and events

torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
    auto output = torch::empty_like(x);
    size_t total = x.numel();
    
    // Define chunk size (number of elements per chunk) for pipelining
    const size_t CHUNK_SIZE = 1 << 20; // 1M elements per chunk

    // For small tensors, use a single kernel launch on the default stream
    if (total <= CHUNK_SIZE) {
        const int threads = 256;
        int blocks = (total + threads - 1) / threads;
        AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "gelu_stream_overlap_simple", ([&] {
            gelu_kernel_compute<scalar_t><<<blocks, threads>>>(x.data_ptr<scalar_t>(),
                                                               output.data_ptr<scalar_t>(),
                                                               total);
        }));
        hipError_t err = hipGetLastError();
        TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));
        return output;
    }

    // Create two CUDA streams: one for memory copies and one for kernel computation
    hipStream_t stream_copy, stream_compute;
    hipError_t err = hipStreamCreate(&stream_copy);
    TORCH_CHECK(err == hipSuccess, "Failed to create stream_copy: ", hipGetErrorString(err));
    err = hipStreamCreate(&stream_compute);
    TORCH_CHECK(err == hipSuccess, "Failed to create stream_compute: ", hipGetErrorString(err));

    // Create events for inter-stream synchronization
    hipEvent_t event_copy_done, event_compute_done;
    err = hipEventCreateWithFlags(&event_copy_done, hipEventDisableTiming);
    TORCH_CHECK(err == hipSuccess, "Failed to create event_copy_done: ", hipGetErrorString(err));
    err = hipEventCreateWithFlags(&event_compute_done, hipEventDisableTiming);
    TORCH_CHECK(err == hipSuccess, "Failed to create event_compute_done: ", hipGetErrorString(err));

    // Allocate double buffers for input and output chunks
    void* d_in[2];
    void* d_out[2];
    size_t buf_bytes = CHUNK_SIZE * x.element_size();
    for (int i = 0; i < 2; i++) {
        err = hipMalloc(&d_in[i], buf_bytes);
        TORCH_CHECK(err == hipSuccess, "Failed to allocate d_in[", i, "]: ", hipGetErrorString(err));
        err = hipMalloc(&d_out[i], buf_bytes);
        TORCH_CHECK(err == hipSuccess, "Failed to allocate d_out[", i, "]: ", hipGetErrorString(err));
    }

    // Get raw pointers for the tensor data
    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "gelu_stream_overlap_pipelined", ([&] {
        auto x_ptr = x.data_ptr<scalar_t>();
        auto y_ptr = output.data_ptr<scalar_t>();

        const int threads = 256;
        size_t offset = 0;
        int chunk_idx = 0;
        while (offset < total) {
            size_t current_chunk = std::min(CHUNK_SIZE, total - offset);
            int buf_idx = chunk_idx % 2;

            // Asynchronously copy input chunk from x to temporary buffer d_in[buf_idx] using stream_copy
            err = hipMemcpyAsync(d_in[buf_idx],
                                  x_ptr + offset,
                                  current_chunk * sizeof(scalar_t),
                                  hipMemcpyDeviceToDevice,
                                  stream_copy);
            TORCH_CHECK(err == hipSuccess, "hipMemcpyAsync (input) failed: ", hipGetErrorString(err));

            // Record event on stream_copy when input copy is complete
            err = hipEventRecord(event_copy_done, stream_copy);
            TORCH_CHECK(err == hipSuccess, "hipEventRecord (copy) failed: ", hipGetErrorString(err));

            // In stream_compute, wait for the input copy to complete
            err = hipStreamWaitEvent(stream_compute, event_copy_done, 0);
            TORCH_CHECK(err == hipSuccess, "hipStreamWaitEvent (copy) failed: ", hipGetErrorString(err));

            // Launch the GELU kernel on the current chunk on stream_compute
            int blocks = (current_chunk + threads - 1) / threads;
            gelu_kernel_compute<scalar_t><<<blocks, threads, 0, stream_compute>>>(
                reinterpret_cast<scalar_t*>(d_in[buf_idx]),
                reinterpret_cast<scalar_t*>(d_out[buf_idx]),
                current_chunk);

            // Record event on stream_compute when computation is done
            err = hipEventRecord(event_compute_done, stream_compute);
            TORCH_CHECK(err == hipSuccess, "hipEventRecord (compute) failed: ", hipGetErrorString(err));

            // In stream_copy, wait for the computation to finish
            err = hipStreamWaitEvent(stream_copy, event_compute_done, 0);
            TORCH_CHECK(err == hipSuccess, "hipStreamWaitEvent (compute) failed: ", hipGetErrorString(err));

            // Asynchronously copy the computed result from d_out[buf_idx] back to the output tensor
            err = hipMemcpyAsync(y_ptr + offset,
                                  d_out[buf_idx],
                                  current_chunk * sizeof(scalar_t),
                                  hipMemcpyDeviceToDevice,
                                  stream_copy);
            TORCH_CHECK(err == hipSuccess, "hipMemcpyAsync (output) failed: ", hipGetErrorString(err));

            offset += current_chunk;
            chunk_idx++;
        }
    }));

    // Synchronize to ensure all operations are complete
    hipStreamSynchronize(stream_copy);
    hipStreamSynchronize(stream_compute);

    // Clean up temporary resources
    for (int i = 0; i < 2; i++) {
        hipFree(d_in[i]);
        hipFree(d_out[i]);
    }
    hipEventDestroy(event_copy_done);
    hipEventDestroy(event_compute_done);
    hipStreamDestroy(stream_copy);
    hipStreamDestroy(stream_compute);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "GELU activation forward with overlapped memory transfers and computation (CUDA)");
}
