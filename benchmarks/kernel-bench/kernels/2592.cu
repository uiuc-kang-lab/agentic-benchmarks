#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

// Kernel that processes a chunk of the input using a grid-stride loop
template <typename scalar_t>
__global__ void relu_kernel_chunk(
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ input,
    const int64_t size) {
    
    // Use 4-element vector types for coalesced memory access when possible
    using Vec4 = typename std::conditional<std::is_same<scalar_t, float>::value,
        float4, double4>::type;
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    // Vector processing for aligned data
    const int vec_size = 4;
    const int64_t vec_elements = size / vec_size;
    const Vec4* input_vec = reinterpret_cast<const Vec4*>(input);
    Vec4* output_vec = reinterpret_cast<Vec4*>(output);
    
    // Process 4 elements at a time
    for (int64_t i = idx; i < vec_elements; i += stride) {
        Vec4 in = input_vec[i];
        if constexpr (std::is_same<scalar_t, float>::value) {
            in.x = (in.x > 0) ? in.x : 0.0f;
            in.y = (in.y > 0) ? in.y : 0.0f;
            in.z = (in.z > 0) ? in.z : 0.0f;
            in.w = (in.w > 0) ? in.w : 0.0f;
        } else {
            in.x = (in.x > 0) ? in.x : 0.0;
            in.y = (in.y > 0) ? in.y : 0.0;
            in.z = (in.z > 0) ? in.z : 0.0;
            in.w = (in.w > 0) ? in.w : 0.0;
        }
        output_vec[i] = in;
    }
    
    // Handle remaining elements
    const int64_t remaining_start = vec_elements * vec_size;
    for (int64_t i = remaining_start + idx; i < size; i += stride) {
        scalar_t x = input[i];
        output[i] = (x > 0) ? x : static_cast<scalar_t>(0);
    }
}

// PyTorch wrapper function that splits the work into chunks processed on separate CUDA streams
torch::Tensor forward(torch::Tensor input) {
    // Get total number of elements
    const int64_t numel = input.numel();
    // Allocate output tensor (uninitialized memory)
    auto output = torch::empty_like(input);

    const int threads = 256;
    // Choose number of streams; for small tensors use a single stream
    int num_streams = 4;
    if (numel < 4096) {
        num_streams = 1;
    }
    // Determine chunk size per stream
    const int64_t chunk_size = (numel + num_streams - 1) / num_streams;

    // Create CUDA streams
    std::vector<hipStream_t> streams(num_streams);
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
    }

    // Launch kernel for each chunk on its own stream
    for (int i = 0; i < num_streams; i++) {
        int64_t offset = i * chunk_size;
        if (offset >= numel) break;
        int64_t current_chunk = std::min(chunk_size, numel - offset);
        int blocks = (current_chunk + threads - 1) / threads;

        AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "relu_kernel_chunk", ([&] {
            relu_kernel_chunk<scalar_t><<<blocks, threads, 0, streams[i]>>>(
                output.data_ptr<scalar_t>() + offset,
                input.data_ptr<scalar_t>() + offset,
                current_chunk);
        }));
    }

    // Synchronize and destroy streams
    for (int i = 0; i < num_streams; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Streamed ReLU forward (CUDA)");
}
