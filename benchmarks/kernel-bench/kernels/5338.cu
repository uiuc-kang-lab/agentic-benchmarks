#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>

struct PoolParams {
    int batch_size;
    int channels;
    int input_height;
    int input_width;
    int output_height;
    int output_width;
    int kernel_size;
    int stride;
    int padding;
    int dilation;
};

__constant__ PoolParams c_params;

template <typename scalar_t, int KERNEL_SIZE>
__global__ void optimized_max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output
) {
    const int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= c_params.batch_size * c_params.channels * 
                      c_params.output_height * c_params.output_width) return;

    const int ow = output_idx % c_params.output_width;
    const int oh = (output_idx / c_params.output_width) % c_params.output_height;
    const int c = (output_idx / (c_params.output_width * c_params.output_height)) % c_params.channels;
    const int b = output_idx / (c_params.output_width * c_params.output_height * c_params.channels);

    const int input_batch_offset = b * (c_params.channels * c_params.input_height * c_params.input_width);
    const int input_channel_offset = c * (c_params.input_height * c_params.input_width);
    
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    const int ih_base = oh * c_params.stride - c_params.padding;
    const int iw_base = ow * c_params.stride - c_params.padding;

    if constexpr (KERNEL_SIZE == 2) {
        if (ih_base >= 0 && ih_base < c_params.input_height) {
            const int ih_offset = ih_base * c_params.input_width;
            if (iw_base >= 0 && iw_base < c_params.input_width) {
                const int idx = input_batch_offset + input_channel_offset + ih_offset + iw_base;
                max_val = __ldg(&input[idx]);
            }
            if (iw_base + c_params.dilation >= 0 && iw_base + c_params.dilation < c_params.input_width) {
                const int idx = input_batch_offset + input_channel_offset + ih_offset + (iw_base + c_params.dilation);
                max_val = max(max_val, __ldg(&input[idx]));
            }
        }
        if (ih_base + c_params.dilation >= 0 && ih_base + c_params.dilation < c_params.input_height) {
            const int ih_offset = (ih_base + c_params.dilation) * c_params.input_width;
            if (iw_base >= 0 && iw_base < c_params.input_width) {
                const int idx = input_batch_offset + input_channel_offset + ih_offset + iw_base;
                max_val = max(max_val, __ldg(&input[idx]));
            }
            if (iw_base + c_params.dilation >= 0 && iw_base + c_params.dilation < c_params.input_width) {
                const int idx = input_batch_offset + input_channel_offset + ih_offset + (iw_base + c_params.dilation);
                max_val = max(max_val, __ldg(&input[idx]));
            }
        }
    }
    else {
        #pragma unroll
        for (int kh = 0; kh < KERNEL_SIZE; kh++) {
            const int ih = ih_base + kh * c_params.dilation;
            if (ih >= 0 && ih < c_params.input_height) {
                const int ih_offset = ih * c_params.input_width;
                #pragma unroll
                for (int kw = 0; kw < KERNEL_SIZE; kw++) {
                    const int iw = iw_base + kw * c_params.dilation;
                    if (iw >= 0 && iw < c_params.input_width) {
                        const int idx = input_batch_offset + input_channel_offset + ih_offset + iw;
                        max_val = max(max_val, __ldg(&input[idx]));
                    }
                }
            }
        }
    }

    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    PoolParams host_params = {
        batch_size, channels, input_height, input_width,
        output_height, output_width, kernel_size, stride, padding, dilation
    };
    hipMemcpyToSymbol(HIP_SYMBOL(c_params), &host_params, sizeof(PoolParams));

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        switch(kernel_size) {
            case 2:
                optimized_max_pool2d_kernel<scalar_t, 2><<<blocks, threads>>>(
                    input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>());
                break;
            case 3:
                optimized_max_pool2d_kernel<scalar_t, 3><<<blocks, threads>>>(
                    input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>());
                break;
            default:
                optimized_max_pool2d_kernel<scalar_t, -1><<<blocks, threads>>>(
                    input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>());
        }
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Optimized Max Pool 2D forward (CUDA)");
}