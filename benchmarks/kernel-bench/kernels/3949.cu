#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void softsign_kernel(const float* x, float* out, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        out[idx] = x[idx] / (1.0f + fabsf(x[idx]));
    }
}

torch::Tensor forward(torch::Tensor x) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int num_elements = x.numel();
    int threads = 1024;
    int blocks = (num_elements + threads - 1) / threads;

    hipStream_t stream;
    hipStreamCreate(&stream);

    float *d_x, *d_out;
    hipMalloc(&d_x, num_elements * sizeof(float));
    hipMalloc(&d_out, num_elements * sizeof(float));

    hipMemcpyAsync(d_x, x.data_ptr<float>(), num_elements * sizeof(float), hipMemcpyHostToDevice, stream);
    
    softsign_kernel<<<blocks, threads, 0, stream>>>(d_x, d_out, num_elements);

    hipMemcpyAsync(out.data_ptr<float>(), d_out, num_elements * sizeof(float), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    hipFree(d_x);
    hipFree(d_out);
    hipStreamDestroy(stream);

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Softsign activation with streams (CUDA)");
}