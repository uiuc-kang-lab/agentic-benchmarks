#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

constexpr int NUM_STREAMS = 4;
constexpr int CHUNK_SIZE = 1024;
constexpr int TILE_SIZE = 32;
constexpr int MAX_THREADS_PER_BLOCK = 256;

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   const int N,
                                   const int chunk_start,
                                   const int chunk_size) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int chunk_offset = chunk_start * N;
    
    const int tx = threadIdx.x % TILE_SIZE;
    const int ty = threadIdx.x / TILE_SIZE;
    
    if (tid < chunk_size * N) {
        const int local_row = tid / N;
        const int global_row = chunk_start + local_row;
        const int col = tid % N;
        
        if (global_row < N && col < N) {
            if (global_row < col) {
                C[chunk_offset + local_row * N + col] = 0.f;
            } else {
                float sum = 0.f;
                
                for (int t = 0; t < (global_row - col + TILE_SIZE - 1) / TILE_SIZE; ++t) {
                    if (tx + t * TILE_SIZE <= global_row) {
                        As[ty][tx] = A[chunk_offset + local_row * N + (t * TILE_SIZE + tx)];
                        Bs[ty][tx] = B[(t * TILE_SIZE + ty) * N + col];
                    }
                    __syncthreads();
                    
                    #pragma unroll 16
                    for (int k = 0; k < TILE_SIZE; ++k) {
                        if (t * TILE_SIZE + k <= global_row && t * TILE_SIZE + k >= col) {
                            sum += As[ty][k] * Bs[k][tx];
                        }
                    }
                    __syncthreads();
                }
                C[chunk_offset + local_row * N + col] = sum;
            }
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    const int N = A.size(0);
    auto C = torch::empty_like(A);
    
    std::vector<hipStream_t> streams(NUM_STREAMS);
    for (int i = 0; i < NUM_STREAMS; ++i) {
        TORCH_CHECK(hipStreamCreate(&streams[i]) == hipSuccess, 
                   "Failed to create CUDA stream");
    }

    for (int chunk_start = 0; chunk_start < N; chunk_start += CHUNK_SIZE) {
        const int stream_idx = (chunk_start / CHUNK_SIZE) % NUM_STREAMS;
        const int current_chunk_size = std::min(CHUNK_SIZE, N - chunk_start);
        const int numElements = current_chunk_size * N;
        const int numBlocks = (numElements + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;

        triangular_mm_kernel<<<numBlocks, MAX_THREADS_PER_BLOCK, 0, streams[stream_idx]>>>(
            A.data_ptr<float>(),
            B.data_ptr<float>(),
            C.data_ptr<float>(),
            N,
            chunk_start,
            current_chunk_size
        );
    }

    for (int i = 0; i < NUM_STREAMS; ++i) {
        TORCH_CHECK(hipStreamSynchronize(streams[i]) == hipSuccess,
                   "Stream synchronization failed");
        TORCH_CHECK(hipStreamDestroy(streams[i]) == hipSuccess,
                   "Stream destruction failed");
    }

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}