#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device function that performs the convolution for one output element.
__device__ __forceinline__ float compute_conv1d(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    int b,
    int oc,
    int o,
    int in_channels,
    int in_size,
    int kernel_size,
    int stride,
    int dilation) {
  float sum = 0.0f;
  int start_pos = o * stride;
  int end_pos = start_pos + (kernel_size - 1) * dilation;

  if (end_pos < in_size) {
    for (int ic = 0; ic < in_channels; ++ic) {
      const float* x_ptr = x + b * (in_channels * in_size) + ic * in_size + start_pos;
      const float* w_ptr = weight + oc * (in_channels * kernel_size) + ic * kernel_size;
      #pragma unroll
      for (int k = 0; k < kernel_size; ++k) {
        sum += x_ptr[k * dilation] * w_ptr[k];
      }
    }
  } else {
    for (int ic = 0; ic < in_channels; ++ic) {
      const float* x_ptr = x + b * (in_channels * in_size) + ic * in_size;
      const float* w_ptr = weight + oc * (in_channels * kernel_size) + ic * kernel_size;
      #pragma unroll
      for (int k = 0; k < kernel_size; ++k) {
        int pos = start_pos + k * dilation;
        if (pos < in_size) {
          sum += x_ptr[pos] * w_ptr[k];
        }
      }
    }
  }
  return sum;
}

// CUDA kernel that computes the 1D convolution using the modular device function above
__global__ void conv1d_streamed_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int B,
    int in_channels,
    int in_size,
    int out_channels,
    int kernel_size,
    int out_size,
    int stride,
    int dilation) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total_elements = B * out_channels * out_size;
  if (idx >= total_elements) return;

  int o = idx % out_size;
  int tmp = idx / out_size;
  int oc = tmp % out_channels;
  int b = tmp / out_channels;

  float sum = compute_conv1d(x, weight, b, oc, o, in_channels, in_size, kernel_size, stride, dilation);

  if (bias != nullptr) {
    sum += bias[oc];
  }

  int out_idx = b * (out_channels * out_size) + oc * out_size + o;
  output[out_idx] = sum;
}

// Forward function exposed via pybind11
// This function uses CUDA streams to overlap memory transfers and computation
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int dilation) {
  TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
  TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
  TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
  TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");
  TORCH_CHECK(x.dim() == 3, "x must be 3D");
  TORCH_CHECK(weight.dim() == 3, "weight must be 3D");
  TORCH_CHECK(weight.size(1) == x.size(1), "Input channels mismatch");

  if (bias.has_value()) {
    TORCH_CHECK(bias.value().device().is_cuda(), "bias must be a CUDA tensor");
    TORCH_CHECK(bias.value().is_contiguous(), "bias must be contiguous");
    TORCH_CHECK(bias.value().dim() == 1, "bias must be 1D");
    TORCH_CHECK(bias.value().size(0) == weight.size(0), "Bias size mismatch");
  }

  int B = x.size(0);
  int in_channels = x.size(1);
  int in_size = x.size(2);
  int out_channels = weight.size(0);
  int kernel_size = weight.size(2);

  int out_size = (in_size - dilation * (kernel_size - 1) - 1) / stride + 1;
  TORCH_CHECK(out_size > 0, "Invalid output size");

  auto output = torch::empty({B, out_channels, out_size}, x.options());
  if (output.numel() == 0) return output;

  const float* x_data = x.data_ptr<float>();
  const float* weight_data = weight.data_ptr<float>();
  const float* bias_data = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
  float* output_data = output.data_ptr<float>();

  int total_elements = B * out_channels * out_size;
  int threads = 256;
  int blocks = (total_elements + threads - 1) / threads;

  hipStream_t stream;
  hipStreamCreate(&stream);

  // Launch kernel in a separate stream
  conv1d_streamed_kernel<<<blocks, threads, 0, stream>>>(
      x_data,
      weight_data,
      bias_data,
      output_data,
      B,
      in_channels,
      in_size,
      out_channels,
      kernel_size,
      out_size,
      stride,
      dilation);

  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  hipError_t err = hipGetLastError();
  TORCH_CHECK(err == hipSuccess, "CUDA kernel launch error: ", hipGetErrorString(err));

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "Streamed 1D convolution forward (CUDA)");
}
