#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <cstdint>

// Define block size based on experimental tuning on the H100 GPU
#define BLOCK_SIZE 512
#define CHUNK_SIZE 4  // Each thread processes 4 float4 elements

// Kernel using vectorized loads (float4) and thread coarsening for better instruction-level parallelism
__global__ void multiplyVectorizedKernel(const float4* __restrict__ A,
                                           float4* __restrict__ C,
                                           float s,
                                           int64_t count) {
    // Use shared memory to cache scalar value
    __shared__ float s_scalar;
    if (threadIdx.x == 0) {
        s_scalar = s;
    }
    __syncthreads();
    
    // Each thread processes multiple elements (thread coarsening)
    int tid = threadIdx.x;
    int stride = blockDim.x;
    int base_idx = blockIdx.x * (blockDim.x * CHUNK_SIZE) + tid;
    
    // Prefetch data into registers for better instruction-level parallelism
    float4 a[CHUNK_SIZE];
    
    #pragma unroll
    for (int i = 0; i < CHUNK_SIZE; i++) {
        int idx = base_idx + i * stride;
        if (idx < count) {
            a[i] = A[idx];
            // Multiply and store using the cached scalar value
            a[i].x *= s_scalar;
            a[i].y *= s_scalar;
            a[i].z *= s_scalar;
            a[i].w *= s_scalar;
            C[idx] = a[i];
        }
    }
}

// Kernel to deal with the remaining elements if the total number is not divisible by 4.
__global__ void multiplyRemainderKernel(const float* __restrict__ A,
                                         float* __restrict__ C,
                                         float s,
                                         int64_t start,
                                         int64_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = start + idx;
    if (i < size) {
        C[i] = A[i] * s;
    }
}

// Forward function that selects the vectorized kernel path if both A and C are 16-byte aligned.
// Using an experiment-based block size (BLOCK_SIZE = 512) to help maximize occupancy on the H100 GPU.

torch::Tensor forward(torch::Tensor A, float s) {
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor.");
    TORCH_CHECK(A.scalar_type() == torch::kFloat, "Input tensor A must be of type float.");

    auto C = torch::empty_like(A);
    int64_t size = A.numel();
    const int threads = BLOCK_SIZE;  // Experimentally tuned block size

    // Check alignment for vectorized memory access (float4 requires 16-byte alignment)
    bool aligned = ((reinterpret_cast<uintptr_t>(A.data_ptr<float>()) % sizeof(float4)) == 0) &&
                   ((reinterpret_cast<uintptr_t>(C.data_ptr<float>()) % sizeof(float4)) == 0);

    if (aligned && size >= 4) {
        int64_t count = size / 4;     // Number of float4 elements
        int remainder = size % 4;       // Remaining elements
        int blocks = (count + threads - 1) / threads;

        multiplyVectorizedKernel<<<blocks, threads>>>(
            reinterpret_cast<const float4*>(A.data_ptr<float>()),
            reinterpret_cast<float4*>(C.data_ptr<float>()),
            s,
            count);

        if (remainder > 0) {
            int start = count * 4;
            int remBlocks = (remainder + threads - 1) / threads;
            multiplyRemainderKernel<<<remBlocks, threads>>>(
                A.data_ptr<float>(),
                C.data_ptr<float>(),
                s,
                start,
                size);
        }
    } else {
        int blocks = (size + threads - 1) / threads;
        multiplyRemainderKernel<<<blocks, threads>>>(A.data_ptr<float>(), C.data_ptr<float>(), s, 0, size);
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Block size optimized matrix-scalar multiplication kernel");
}
