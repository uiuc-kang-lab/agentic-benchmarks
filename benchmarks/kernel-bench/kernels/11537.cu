#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device functions for modular kernel components
__device__ __forceinline__ float compute_kl_div(float log_pred, float target) {
    return __expf(log_pred) - target * log_pred;
}

__device__ __forceinline__ float warp_reduce_sum(float val) {
    #pragma unroll
    for (int offset = 32/2; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__global__ void optimized_modular_kl_div_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    // Calculate aligned index for coalesced memory access
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int num_threads = blockDim.x;
    const int warp_size = 32;
    const int warp_id = tid / warp_size;
    const int lane_id = tid % warp_size;
    const int aligned_idx = bid * num_threads + warp_id * warp_size + lane_id;
    
    // Shared memory for partial sums - only need warp-level storage
    extern __shared__ float partial_sums[];
    
    // Local accumulator for better register utilization
    float thread_sum = 0.0f;
    
    // Process elements with aligned access and grid stride
    for (int i = aligned_idx; i < n; i += gridDim.x * num_threads) {
        if (i < n) {
            thread_sum += compute_kl_div(log_predictions[i], targets[i]);
        }
    }
    
    // Warp-level reduction using shuffle operations
    thread_sum = warp_reduce_sum(thread_sum);
    
    // Store warp results in shared memory
    if (lane_id == 0) {
        partial_sums[warp_id] = thread_sum;
    }
    __syncthreads();
    
    // Final reduction by first warp
    if (warp_id == 0) {
        float sum = (tid < blockDim.x/warp_size) ? partial_sums[lane_id] : 0.0f;
        sum = warp_reduce_sum(sum);
        
        if (lane_id == 0) {
            atomicAdd(output, sum);
        }
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    // Optimized launch parameters
    const int threads = 256; // Multiple of warp size
    const int blocks = min((n + threads - 1) / threads, 1024);
    const int shared_mem = (threads/32) * sizeof(float);
    
    optimized_modular_kl_div_kernel<<<blocks, threads, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward (CUDA)");
}