#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Constant memory for frequently accessed parameters
__constant__ int c_kernel_size;
__constant__ int c_stride;
__constant__ int c_padding;

__global__ void avg_pool3d_tuned_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int batch_size, int channels,
    int in_d, int in_h, int in_w,
    int out_d, int out_h, int out_w) {
    
    // 2D block configuration for better occupancy
    const int tid_x = threadIdx.x;
    const int tid_y = threadIdx.y;
    const int idx_x = blockIdx.x * blockDim.x + tid_x;
    const int idx_y = blockIdx.y * blockDim.y + tid_y;
    
    // Calculate batch and channel indices
    const int batch_channel_idx = blockIdx.z;
    const int n = batch_channel_idx / channels;
    const int c = batch_channel_idx % channels;
    
    // Early exit conditions
    if (idx_x >= out_w || idx_y >= out_h || n >= batch_size)
        return;
    
    // Process multiple depth slices per thread block
    for (int d_out = 0; d_out < out_d; d_out++) {
        // Calculate window boundaries
        const int d_start = d_out * c_stride - c_padding;
        const int h_start = idx_y * c_stride - c_padding;
        const int w_start = idx_x * c_stride - c_padding;
        
        // Calculate clamped boundaries
        const int d_start_clamped = max(0, d_start);
        const int h_start_clamped = max(0, h_start);
        const int w_start_clamped = max(0, w_start);
        const int d_end = min(d_start + c_kernel_size, in_d);
        const int h_end = min(h_start + c_kernel_size, in_h);
        const int w_end = min(w_start + c_kernel_size, in_w);
        
        float sum = 0.0f;
        const int base_idx = ((n * channels + c) * in_d);
        
        #pragma unroll 3
        for (int d = d_start_clamped; d < d_end; ++d) {
            const int d_offset = (base_idx + d) * in_h;
            #pragma unroll 3
            for (int h = h_start_clamped; h < h_end; ++h) {
                const int h_offset = (d_offset + h) * in_w;
                #pragma unroll 3
                for (int w = w_start_clamped; w < w_end; ++w) {
                    sum += __ldg(&input[h_offset + w]);
                }
            }
        }
        
        const float pool_size = static_cast<float>(c_kernel_size * c_kernel_size * c_kernel_size);
        const int out_idx = (((n * channels + c) * out_d + d_out) * out_h + idx_y) * out_w + idx_x;
        output[out_idx] = sum / pool_size;
    }
}

at::Tensor forward(at::Tensor input, int kernel_size, int stride, int padding) {
    TORCH_CHECK(input.dim() == 5, "Input tensor must be 5-dimensional");
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");
    
    const int batch_size = input.size(0);
    const int channels = input.size(1);
    const int in_d = input.size(2);
    const int in_h = input.size(3);
    const int in_w = input.size(4);
    
    const int out_d = (in_d + 2 * padding - kernel_size) / stride + 1;
    const int out_h = (in_h + 2 * padding - kernel_size) / stride + 1;
    const int out_w = (in_w + 2 * padding - kernel_size) / stride + 1;
    
    // Copy constants to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(c_kernel_size), &kernel_size, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(c_stride), &stride, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(c_padding), &padding, sizeof(int));
    
    auto output = at::empty({batch_size, channels, out_d, out_h, out_w}, input.options());
    
    // Optimized block configuration for H100
    dim3 threads(32, 16);  // 512 threads per block
    dim3 blocks(
        (out_w + threads.x - 1) / threads.x,
        (out_h + threads.y - 1) / threads.y,
        batch_size * channels
    );
    
    avg_pool3d_tuned_kernel<<<blocks, threads>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size, channels,
        in_d, in_h, in_w,
        out_d, out_h, out_w
    );
    
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed : ", hipGetErrorString(err));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D Average Pooling forward (CUDA)");
}