#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Kernel that uses a grid-stride loop to handle workloads larger than available threads
__global__ void leaky_relu_kernel_stride(const float* __restrict__ x, float* __restrict__ out, float negative_slope, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < n; i += stride) {
        float val = x[i];
        out[i] = (val > 0.0f) ? val : val * negative_slope;
    }
}

// Forward function that sets up and launches the kernel
torch::Tensor leaky_relu_forward(torch::Tensor x, float negative_slope) {
    CHECK_INPUT(x);
    auto out = torch::empty_like(x);
    int n = x.numel();

    // Use a fixed kernel configuration with a grid-stride loop
    const int threads = 1024;
    const int blocks = 1024;  // Using 1024 blocks to cover large workloads

    leaky_relu_kernel_stride<<<blocks, threads>>>(x.data_ptr<float>(), out.data_ptr<float>(), negative_slope, n);

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &leaky_relu_forward, "LeakyReLU forward with grid-stride loop (CUDA)");
}
