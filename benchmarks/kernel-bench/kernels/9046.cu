#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

// Define maximum allowed sizes for constant memory storage
#define MAX_WEIGHT_SIZE 15360
#define MAX_BIAS_SIZE 1024

// Declare constant memory arrays for weight and bias
__constant__ float c_weight[MAX_WEIGHT_SIZE];
__constant__ float c_bias[MAX_BIAS_SIZE];

// CUDA kernel that uses constant memory for weight and bias with batch offset support
__global__ void conv1d_const_stream_kernel(
    const float* __restrict__ x,
    float* output,
    int batch_offset,  // offset in the batch dimension
    int B,             // number of batches in this kernel launch
    int in_channels,
    int in_size,
    int out_channels,
    int kernel_size,
    int out_size,
    int stride,
    int dilation,
    bool use_bias
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = B * out_channels * out_size;  // local number of output elements
    if (idx >= total_elements) return;

    int o = idx % out_size;
    idx /= out_size;
    int oc = idx % out_channels;
    int local_b = idx / out_channels;
    int b = batch_offset + local_b;

    float sum = 0.0f;
    int base_weight_oc = oc * (in_channels * kernel_size);
    for (int ic = 0; ic < in_channels; ++ic) {
        int x_base = b * (in_channels * in_size) + ic * in_size;
        int weight_base = base_weight_oc + ic * kernel_size;
        for (int k = 0; k < kernel_size; ++k) {
            int input_pos = o * stride + k * dilation;
            // Since out_size is computed to ensure valid convolution, we remove the boundary check
            int x_idx = x_base + input_pos;
            int w_idx = weight_base + k;
            sum += x[x_idx] * c_weight[w_idx];
        }
    }
    if (use_bias) {
        sum += c_bias[oc];
    }

    int out_idx = b * (out_channels * out_size) + oc * out_size + o;
    output[out_idx] = sum;
}

// Forward function combining constant memory and pipelined streams
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int dilation,
    int num_streams = 4  // default number of CUDA streams to use
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");
    TORCH_CHECK(x.dim() == 3, "x must be 3D (B, in_channels, in_size)");
    TORCH_CHECK(weight.dim() == 3, "weight must be 3D (out_channels, in_channels, kernel_size)");
    TORCH_CHECK(weight.size(1) == x.size(1), "Input channel mismatch between x and weight");

    bool use_bias = false;
    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        TORCH_CHECK(bias->dim() == 1, "bias must be 1D");
        TORCH_CHECK(bias->size(0) == weight.size(0), "Bias size must match number of output channels");
        use_bias = true;
    }

    int B_total = x.size(0);
    int in_channels = x.size(1);
    int in_size = x.size(2);
    int out_channels = weight.size(0);
    int kernel_size = weight.size(2);
    int out_size = (in_size - dilation * (kernel_size - 1) - 1) / stride + 1;
    TORCH_CHECK(out_size > 0, "Invalid output size computed");

    // Ensure weight (and bias) fit into constant memory
    TORCH_CHECK(weight.numel() <= MAX_WEIGHT_SIZE, "Weight tensor too large for constant memory");
    if (use_bias) {
        TORCH_CHECK(bias->numel() <= MAX_BIAS_SIZE, "Bias tensor too large for constant memory");
    }

    // Copy weight (and bias) into constant memory
    size_t weight_bytes = weight.numel() * sizeof(float);
    hipMemcpyToSymbol(HIP_SYMBOL(c_weight), weight.data_ptr<float>(), weight_bytes, 0, hipMemcpyDeviceToDevice);
    if (use_bias) {
        size_t bias_bytes = bias->numel() * sizeof(float);
        hipMemcpyToSymbol(HIP_SYMBOL(c_bias), bias->data_ptr<float>(), bias_bytes, 0, hipMemcpyDeviceToDevice);
    }

    auto output = torch::empty({B_total, out_channels, out_size}, x.options());
    if (output.numel() == 0) return output;

    const float* x_data = x.data_ptr<float>();
    float* output_data = output.data_ptr<float>();

    // Set up CUDA streams for pipelined execution over batch segments
    int effective_streams = std::min(num_streams, B_total);
    std::vector<hipStream_t> streams(effective_streams);
    for (int i = 0; i < effective_streams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    // Partition the batch dimension among the available streams
    int batch_per_stream = (B_total + effective_streams - 1) / effective_streams;
    int threads = 256;
    for (int i = 0; i < effective_streams; ++i) {
        int start_B = i * batch_per_stream;
        int end_B = std::min(start_B + batch_per_stream, B_total);
        if (start_B >= end_B) continue;
        int current_B = end_B - start_B;
        int total_elements_segment = current_B * out_channels * out_size;
        int blocks = (total_elements_segment + threads - 1) / threads;

        conv1d_const_stream_kernel<<<blocks, threads, 0, streams[i]>>>(
            x_data,
            output_data,
            start_B,
            current_B,
            in_channels,
            in_size,
            out_channels,
            kernel_size,
            out_size,
            stride,
            dilation,
            use_bias
        );
    }

    // Synchronize and clean up streams
    for (int i = 0; i < effective_streams; ++i) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel launch error: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "1D convolution forward with constant memory and CUDA streams (CUDA)",
          pybind11::arg("x"),
          pybind11::arg("weight"),
          pybind11::arg("bias") = torch::Tensor(),
          pybind11::arg("stride"),
          pybind11::arg("dilation"),
          pybind11::arg("num_streams") = 4);
}
