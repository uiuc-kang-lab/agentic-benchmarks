#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Device helper: define an inline exponential function for float and double.
template <typename scalar_t>
__device__ inline scalar_t my_exp(scalar_t x);

template <>
__device__ inline float my_exp<float>(float x) {
    return expf(x);
}

template <>
__device__ inline double my_exp<double>(double x) {
    return exp(x);
}

// CUDA kernel that applies the SELU activation in an elementwise manner.
// Atomic operations are intentionally omitted because each thread processes independent data,
// eliminating race conditions and avoiding unnecessary global memory contention.
// If an atomic update were required (e.g., for a reduction), a block-level reduction would be used
// to minimize atomic operations to one per block.

template <typename scalar_t>
__global__ void selu_kernel_no_atomic(const scalar_t* __restrict__ input,
                                        scalar_t* __restrict__ output,
                                        size_t numel) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = gridDim.x * blockDim.x;
    for (size_t i = idx; i < numel; i += stride) {
        scalar_t x = __ldg(&input[i]);
        scalar_t res = (x > static_cast<scalar_t>(0))
                           ? x
                           : static_cast<scalar_t>(1.67326324235437728481) * (my_exp(x) - static_cast<scalar_t>(1));
        output[i] = static_cast<scalar_t>(1.05070098735548049342) * res;
    }
}

// Host function that launches the SELU kernel without any unnecessary atomic operations.
// This ensures that each thread writes to a unique output location, thereby avoiding race conditions.

torch::Tensor selu_forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    auto output = torch::empty_like(input);
    size_t numel = input.numel();
    const int threads = 1024;
    const int blocks = (numel + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "selu_no_atomic_cuda", ([&] {
        const scalar_t* input_ptr = input.data_ptr<scalar_t>();
        scalar_t* output_ptr = output.data_ptr<scalar_t>();
        selu_kernel_no_atomic<scalar_t><<<blocks, threads>>>(input_ptr, output_ptr, numel);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &selu_forward, "SELU Activation Forward without unnecessary atomic operations (CUDA)");
}
