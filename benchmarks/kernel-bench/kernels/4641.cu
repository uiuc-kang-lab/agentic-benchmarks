#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel function for RMS normalization
template <typename scalar_t>
__global__ void rms_norm_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int num_features,
    const int numel_per_batch,
    const float eps
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int total_elements = batch_size * numel_per_batch;
    
    for (int idx = tid; idx < total_elements; idx += stride) {
        const int batch_id = idx / numel_per_batch;
        const int offset_in_batch = idx % numel_per_batch;
        const int batch_offset = batch_id * num_features * numel_per_batch;
        
        // Calculate sum of squares using grid-stride loop
        scalar_t sumsq = 0.0f;
        #pragma unroll
        for (int feat = 0; feat < num_features; feat++) {
            const scalar_t val = input[batch_offset + feat * numel_per_batch + offset_in_batch];
            sumsq += val * val;
        }
        
        // Calculate RMS
        const scalar_t rms = sqrt(sumsq / num_features + eps);
        
        // Normalize using grid-stride loop
        #pragma unroll
        for (int feat = 0; feat < num_features; feat++) {
            const int out_idx = batch_offset + feat * numel_per_batch + offset_in_batch;
            output[out_idx] = input[out_idx] / rms;
        }
    }
}

// Host function to launch the kernel with streams
torch::Tensor rms_norm_cuda_forward(torch::Tensor input, float eps) {
    auto output = torch::empty_like(input);
    
    const int batch_size = input.size(0);
    const int num_features = input.size(1);
    
    int numel_per_batch = 1;
    for(int i = 2; i < input.dim(); i++) {
        numel_per_batch *= input.size(i);
    }

    const int threads_per_block = 256;
    const int max_blocks = 65535;
    const int total_elements = batch_size * numel_per_batch;
    const int blocks = min(max_blocks, (total_elements + threads_per_block - 1) / threads_per_block);

    // Create CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "rms_norm_cuda", ([&] {
        rms_norm_kernel<scalar_t><<<blocks, threads_per_block, 0, stream>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            num_features,
            numel_per_batch,
            eps
        );
    }));

    // Synchronize the stream
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &rms_norm_cuda_forward, "RMS normalization forward (CUDA)");
}