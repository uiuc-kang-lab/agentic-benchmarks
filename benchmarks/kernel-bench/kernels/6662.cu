#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void unrolled_coalesced_max_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t outer_size,
    const int64_t dim_size,
    const int64_t inner_size
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_elements = outer_size * inner_size;
    
    if (idx >= total_elements) return;
    
    const int outer_idx = idx / inner_size;
    const int inner_idx = idx % inner_size;
    
    const int64_t start = outer_idx * dim_size * inner_size + inner_idx;
    scalar_t max_val = input[start];

    #pragma unroll 4
    for (int i = 1; i < dim_size; ++i) {
        scalar_t val = input[start + i * inner_size];
        if (val > max_val) max_val = val;
    }
    
    output[idx] = max_val;
}

torch::Tensor max_reduce_cuda_forward(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();
    
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++)
        outer_size *= input.size(i);
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < input.dim(); i++)
        inner_size *= input.size(i);
    
    auto output_sizes = input.sizes().vec();
    output_sizes.erase(output_sizes.begin() + dim);
    auto output = torch::empty(output_sizes, input.options());

    const int threads = 256;
    const int blocks = (outer_size * inner_size + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "max_reduce_forward", ([&] {
        unrolled_coalesced_max_reduce_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            input.size(dim),
            inner_size
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_reduce_cuda_forward, "Max reduce forward (CUDA)");
}