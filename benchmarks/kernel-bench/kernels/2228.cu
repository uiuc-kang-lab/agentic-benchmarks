#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

// Define warp and tile dimensions
#define WARP_SIZE 32
#define TILE_M 8    // Number of output rows computed per block
#define TILE_N 8    // Number of output columns computed per block

// CUDA kernel: Computes C = A.T * B using warp-level parallelism
// A: shape (K, M) stored in row-major order (A[k][i] -> A[k*M + i])
// B: shape (K, N) stored in row-major order (B[k][j] -> B[k*N + j])
// C: shape (M, N) stored in row-major order (C[i][j] -> C[i*N + j])
__global__ void warpDotKernel(const float* __restrict__ A,
                              const float* __restrict__ B,
                              float* __restrict__ C,
                              int K, int M, int N) {
    // Block handles TILE_M rows and TILE_N columns of output
    // Each warp processes one element in the output tile
    const unsigned int warp_size = WARP_SIZE;
    const unsigned int lane = threadIdx.x % warp_size;
    const unsigned int warp_id = threadIdx.x / warp_size;
    
    // Calculate output position
    const unsigned int row = blockIdx.y * TILE_M + (warp_id / TILE_N);
    const unsigned int col = blockIdx.x * TILE_N + (warp_id % TILE_N);
    
    // Initialize accumulator
    float sum = 0.0f;
    
    // Only compute if within bounds
    if (row < M && col < N) {
        // Each thread in warp handles part of reduction
        #pragma unroll 4
        for (int k = lane; k < K; k += warp_size) {
            const float a_val = __ldg(&A[k * M + row]);  // Load from A.T position
            const float b_val = __ldg(&B[k * N + col]);
            sum += a_val * b_val;
        }
        
        // Warp-level reduction using shuffle
        #pragma unroll
        for (int offset = warp_size/2; offset > 0; offset >>= 1) {
            sum += __shfl_xor_sync(0xffffffff, sum, offset);
        }
        
        // First thread in warp writes result
        if (lane == 0) {
            C[row * N + col] = sum;
        }
    }
}

// The forward function exposed via PyBind11.
// Inputs:
//   A: Tensor of shape (K, M) (CUDA, float32)
//   B: Tensor of shape (K, N) (CUDA, float32)
// Returns:
//   C: Tensor of shape (M, N) computed as A.T * B.
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "Input A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input B must be a CUDA tensor");
    TORCH_CHECK(A.dtype() == torch::kFloat32, "Input A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "Input B must be float32");

    int K = A.size(0);
    int M = A.size(1);
    TORCH_CHECK(B.size(0) == K, "Dimension mismatch: A and B must have the same first dimension (K)");
    int N = B.size(1);

    // Allocate output tensor C of shape (M, N)
    auto C = torch::zeros({M, N}, torch::device(A.device()).dtype(A.dtype()));

    // Define grid and block dimensions.
    // Each block computes a tile of size (TILE_M x TILE_N) elements of C.
    // Each element of C is computed by one warp of WARP_SIZE threads.
    dim3 blockDim(TILE_N * WARP_SIZE, TILE_M);
    dim3 grid((N + TILE_N - 1) / TILE_N, (M + TILE_M - 1) / TILE_M);

    const float* A_ptr = A.data_ptr<float>();
    const float* B_ptr = B.data_ptr<float>();
    float* C_ptr = C.data_ptr<float>();

    warpDotKernel<<<grid, blockDim>>>(A_ptr, B_ptr, C_ptr, K, M, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Compute C = A.T * B using warp-level primitives (CUDA)");
}
