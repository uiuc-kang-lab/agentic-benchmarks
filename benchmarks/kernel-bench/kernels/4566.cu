#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/cuda/HIPContext.h>
#include <vector>

#define BLOCK_SIZE_STATS 256
#define BLOCK_SIZE_NORM 256
#define WARP_SIZE 32

template <typename scalar_t, int BLOCK_SIZE>
__global__ void compute_stats_kernel_reduced_sync(
    const scalar_t* __restrict__ x,
    const int N,
    const int C,
    const int spatial,
    const int channels_per_group,
    const int num_groups,
    scalar_t* __restrict__ mean,
    scalar_t* __restrict__ var) {

  const int idx = blockIdx.x;
  const int n = idx / num_groups;
  const int g = idx % num_groups;
  
  const int group_offset = n * C * spatial + g * channels_per_group * spatial;
  const int group_elems = channels_per_group * spatial;

  // Each thread accumulates its portion
  scalar_t sum = 0;
  scalar_t sum_sq = 0;
  
  #pragma unroll 4
  for (int i = threadIdx.x; i < group_elems; i += BLOCK_SIZE) {
    int c = i / spatial;
    int j = i % spatial;
    scalar_t val = x[group_offset + c * spatial + j];
    sum += val;
    sum_sq += val * val;
  }

  // Shared memory for reduction
  extern __shared__ char smem[];
  scalar_t* s_sum = reinterpret_cast<scalar_t*>(smem);
  scalar_t* s_sum_sq = s_sum + BLOCK_SIZE;
  
  s_sum[threadIdx.x] = sum;
  s_sum_sq[threadIdx.x] = sum_sq;
  __syncthreads();  // Single sync point for shared memory writes

  // Warp-level reduction first
  #pragma unroll
  for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
    if (threadIdx.x < offset) {
      s_sum[threadIdx.x] += s_sum[threadIdx.x + offset];
      s_sum_sq[threadIdx.x] += s_sum_sq[threadIdx.x + offset];
    }
  }

  // Block-level reduction with minimal syncs
  if (BLOCK_SIZE > WARP_SIZE) {
    __syncthreads();  // Sync only if we need cross-warp communication
    
    if (threadIdx.x < WARP_SIZE) {
      for (int offset = WARP_SIZE; offset < BLOCK_SIZE; offset += WARP_SIZE) {
        if (threadIdx.x + offset < BLOCK_SIZE) {
          s_sum[threadIdx.x] += s_sum[threadIdx.x + offset];
          s_sum_sq[threadIdx.x] += s_sum_sq[threadIdx.x + offset];
        }
      }
    }
  }

  // First thread writes results
  if (threadIdx.x == 0) {
    scalar_t group_mean = s_sum[0] / group_elems;
    scalar_t group_var = s_sum_sq[0] / group_elems - group_mean * group_mean;
    int out_index = n * num_groups + g;
    mean[out_index] = group_mean;
    var[out_index] = group_var;
  }
}

template <typename scalar_t, int BLOCK_SIZE>
__global__ void group_norm_forward_kernel_reduced_sync(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ mean,
    const scalar_t* __restrict__ var,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    const int N,
    const int C,
    const int spatial,
    const int channels_per_group,
    const int num_groups,
    const scalar_t eps,
    scalar_t* __restrict__ y) {

  const int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  const int total = N * C * spatial;
  if (index >= total) return;

  const int j = index % spatial;
  const int temp = index / spatial;
  const int c = temp % C;
  const int n = temp / C;
  
  const int g = c / channels_per_group;
  const int stats_index = n * num_groups + g;
  
  const scalar_t m = mean[stats_index];
  const scalar_t v = var[stats_index];
  const scalar_t inv_std = rsqrt(v + eps);
  const scalar_t x_val = x[index];
  y[index] = (x_val - m) * inv_std * weight[c] + bias[c];
}

torch::Tensor group_norm_forward_reduced_sync(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    int64_t num_groups,
    double eps) {

  const int N = x.size(0);
  const int C = x.size(1);
  int spatial = 1;
  for (int i = 2; i < x.dim(); i++) {
    spatial *= x.size(i);
  }
  const int channels_per_group = C / num_groups;

  auto y = torch::empty_like(x);
  auto options = torch::TensorOptions().device(x.device()).dtype(x.dtype());
  auto mean = torch::empty({N, num_groups}, options);
  auto var = torch::empty({N, num_groups}, options);

  const int total_groups = N * num_groups;
  const int total_elements = N * C * spatial;
  const int blocks_norm = (total_elements + BLOCK_SIZE_NORM - 1) / BLOCK_SIZE_NORM;

  hipStream_t stream = c10::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "group_norm_forward_reduced_sync", ([&] {
    const size_t shared_mem_size = BLOCK_SIZE_STATS * 2 * sizeof(scalar_t);

    compute_stats_kernel_reduced_sync<scalar_t, BLOCK_SIZE_STATS><<<
        total_groups, BLOCK_SIZE_STATS, shared_mem_size, stream>>>(
        x.data_ptr<scalar_t>(),
        N, C, spatial,
        channels_per_group,
        num_groups,
        mean.data_ptr<scalar_t>(),
        var.data_ptr<scalar_t>());

    group_norm_forward_kernel_reduced_sync<scalar_t, BLOCK_SIZE_NORM><<<
        blocks_norm, BLOCK_SIZE_NORM, 0, stream>>>(
        x.data_ptr<scalar_t>(),
        mean.data_ptr<scalar_t>(),
        var.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(),
        bias.data_ptr<scalar_t>(),
        N, C, spatial,
        channels_per_group,
        num_groups,
        static_cast<scalar_t>(eps),
        y.data_ptr<scalar_t>());
  }));

  return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &group_norm_forward_reduced_sync, "Reduced Sync Group Normalization forward (CUDA)");
}