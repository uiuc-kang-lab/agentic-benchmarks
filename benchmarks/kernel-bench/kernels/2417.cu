#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t, int BLOCK_SIZE>
__global__ void matmul_transpose_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ C,
    const int M,
    const int N,
    const int K) {
    
    __shared__ scalar_t As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ scalar_t Bs[BLOCK_SIZE][BLOCK_SIZE];
    
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    scalar_t sum = 0;
    
    for (int tile = 0; tile < (K + BLOCK_SIZE - 1) / BLOCK_SIZE; ++tile) {
        // Load tiles into shared memory
        if (tile * BLOCK_SIZE + ty < K && row < M)
            As[tx][ty] = A[(tile * BLOCK_SIZE + ty) * M + row];
        else
            As[tx][ty] = 0;
            
        if (tile * BLOCK_SIZE + tx < K && col < N)
            Bs[ty][tx] = B[col * K + tile * BLOCK_SIZE + tx];
        else
            Bs[ty][tx] = 0;
            
        __syncthreads();
        
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            sum += As[tx][k] * Bs[ty][k];
        }
        
        __syncthreads();
    }
    
    // Warp-level reduction within each warp
    #pragma unroll
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }
    
    // Write result
    if (row < M && col < N) {
        if (threadIdx.x % warpSize == 0) {
            C[row * N + col] = sum;
        }
    }
}

torch::Tensor matmul_transpose_cuda(torch::Tensor A, torch::Tensor B) {
    const int K = A.size(0);
    const int M = A.size(1);
    const int N = B.size(0);
    
    auto C = torch::empty({M, N}, A.options());
    
    constexpr int BLOCK_SIZE = 32;
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((M + BLOCK_SIZE - 1) / BLOCK_SIZE,
                (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    AT_DISPATCH_FLOATING_TYPES(A.type(), "matmul_transpose_kernel", ([&] {
        matmul_transpose_kernel<scalar_t, BLOCK_SIZE><<<blocks, threads>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, N, K
        );
    }));
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_transpose_cuda, "Matrix multiplication with transpose forward (CUDA)");
}