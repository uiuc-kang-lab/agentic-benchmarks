#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

// Optimized CUDA kernel for matrix multiplication using __ldg() for read-only global memory loads
// and assuming that the input matrices A and B are allocated on 128-bit boundaries.

template <typename scalar_t>
__global__ void matmul_cuda_kernel(const scalar_t* __restrict__ A, const scalar_t* __restrict__ B,
                                     scalar_t* __restrict__ C, int M, int K, int N) {
    __shared__ scalar_t sA[TILE_WIDTH][TILE_WIDTH];
    __shared__ scalar_t sB[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y; // Index in M dimension
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x; // Index in N dimension

    scalar_t value = 0;
    int num_tiles = (K + TILE_WIDTH - 1) / TILE_WIDTH;

    for (int t = 0; t < num_tiles; ++t) {
        int tiledA_col = t * TILE_WIDTH + threadIdx.x;
        int tiledB_row = t * TILE_WIDTH + threadIdx.y;

        // Use __ldg() for read-only accesses, assuming A and B are 128-bit aligned
        if (row < M && tiledA_col < K)
            sA[threadIdx.y][threadIdx.x] = __ldg(&A[row * K + tiledA_col]);
        else
            sA[threadIdx.y][threadIdx.x] = 0;

        if (col < N && tiledB_row < K)
            sB[threadIdx.y][threadIdx.x] = __ldg(&B[tiledB_row * N + col]);
        else
            sB[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();

        #pragma unroll
        for (int i = 0; i < TILE_WIDTH; ++i) {
            value += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        C[row * N + col] = value;
    }
}

// Forward function

torch::Tensor module_fn(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input tensor B must be a CUDA tensor");

    int64_t M = A.size(0);
    int64_t K = A.size(1);
    int64_t N = B.size(1);
    TORCH_CHECK(K == B.size(0), "Inner dimensions of A and B must match");

    // Allocate output tensor
    auto C = torch::empty({M, N}, A.options());

    dim3 threads_per_block(TILE_WIDTH, TILE_WIDTH);
    dim3 num_blocks((N + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matmul_cuda_kernel", ([&] {
        matmul_cuda_kernel<scalar_t><<<num_blocks, threads_per_block>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, K, N);
    }));

    hipDeviceSynchronize();
    return C;
}

// Pybind11 module binding
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &module_fn, "Optimized matrix multiplication forward (CUDA, __ldg and 128-bit aligned loads)");
}
