#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__constant__ float const_log_predictions[1024]; // Assuming maximum size that fits within constant memory

// CUDA kernel for KL divergence calculation 
__global__ void kl_div_kernel_constant_memory(
    const float* targets, 
    float* output,
    const int n) {
    
    // Get global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Shared memory for partial sums
    extern __shared__ float partial_sums[];
    
    float sum = 0.0f;
    
    // Calculate KL divergence for this thread's elements
    while (idx < n) {
        // F.kl_div implementation:
        // output = exp(log_predictions) - targets * log_predictions
        float log_pred = const_log_predictions[idx];
        float target = targets[idx];
        sum += expf(log_pred) - target * log_pred;
        
        idx += blockDim.x * gridDim.x;
    }
    
    // Store in shared memory
    partial_sums[threadIdx.x] = sum;
    __syncthreads();
    
    // Parallel reduction in shared memory
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            partial_sums[threadIdx.x] += partial_sums[threadIdx.x + stride];
        }
        __syncthreads();
    }
    
    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        atomicAdd(output, partial_sums[0]);
    }
}

torch::Tensor kl_div_cuda_forward_constant_memory(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    // Get tensor sizes
    const int n = log_predictions.numel();
    
    // Create output tensor
    auto output = torch::zeros({1}, log_predictions.options());
    
    // Copy log_predictions to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(const_log_predictions), log_predictions.data_ptr<float>(), n * sizeof(float));
    
    // Launch parameters
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;
    const int shared_mem = threads * sizeof(float);
    
    // Launch kernel
    kl_div_kernel_constant_memory<<<blocks, threads, shared_mem>>>(
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward_constant_memory, "KL divergence forward with constant memory (CUDA)");
}