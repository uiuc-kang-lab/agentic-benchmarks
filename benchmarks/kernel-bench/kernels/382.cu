#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Define constant memory for vector B
__constant__ float B_const[1024];

// CUDA kernel for matrix-vector multiplication using constant memory
template <typename scalar_t>
__global__ void matvec_mul_kernel_const(
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> A,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> C,
    int64_t M,
    int64_t K)
{
    int64_t row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M)
    {
        scalar_t sum = 0;
        for (int64_t k = 0; k < K; ++k)
        {
            sum += A[row][k] * B_const[k];
        }
        C[row][0] = sum;
    }
}

// C++ function that wraps the CUDA kernel
torch::Tensor matvec_mul_cuda_const(torch::Tensor A, torch::Tensor B)
{
    // Ensure input tensors are on CUDA
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");

    // Ensure inputs are contiguous
    A = A.contiguous();
    B = B.contiguous();

    // Get dimensions
    int64_t M = A.size(0);
    int64_t K = A.size(1);

    // Check dimensions
    TORCH_CHECK(B.numel() == K, "B must have the same number of elements as columns in A");
    TORCH_CHECK(B.dim() == 1 || (B.dim() == 2 && B.size(1) == 1), "B must be a vector of shape (K,) or (K, 1)");

    // Flatten B to be a 1D tensor
    auto B_flat = B.view({-1});

    // Copy B to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(B_const), B_flat.data_ptr<scalar_t>(), K * sizeof(scalar_t));

    // Allocate output tensor
    auto C = torch::zeros({M, 1}, A.options());

    // Define block and grid sizes
    int threads = 256;
    int blocks = (M + threads - 1) / threads;

    // Dispatch based on data type
    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matvec_mul_cuda_const", ([&] {
        matvec_mul_kernel_const<scalar_t><<<blocks, threads>>>(
            A.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
            C.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
            M,
            K);
    }));

    // Ensure synchronization
    hipDeviceSynchronize();

    return C;
}

// PyBind11 binding code
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matvec_mul_cuda_const, "Matrix-Vector Multiplication with Constant Memory (CUDA)");
}