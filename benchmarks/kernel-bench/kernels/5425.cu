#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>

// Kernel using grid-stride loop for even workload distribution
// and optimized block size for better load balancing

template <typename scalar_t>
__global__ void max_pool2d_kernel_optimized(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    int total_elements = batch_size * channels * output_height * output_width;
    // Use grid-stride loop to evenly distribute work
    for (int index = blockIdx.x * blockDim.x + threadIdx.x;
         index < total_elements;
         index += gridDim.x * blockDim.x) {

        int ow = index % output_width;
        int temp = index / output_width;
        int oh = temp % output_height;
        temp /= output_height;
        int c = temp % channels;
        int b = temp / channels;

        scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
        int base_h = oh * stride - padding;
        int base_w = ow * stride - padding;
        
        for (int kh = 0; kh < kernel_size; kh++) {
            int ih = base_h + kh * dilation;
            if (ih < 0 || ih >= input_height) continue;
            for (int kw = 0; kw < kernel_size; kw++) {
                int iw = base_w + kw * dilation;
                if (iw < 0 || iw >= input_width) continue;
                int input_index = ((b * channels + c) * input_height + ih) * input_width + iw;
                max_val = max(max_val, input[input_index]);
            }
        }
        output[index] = max_val;
    }
}

// Host function to launch the CUDA kernel
torch::Tensor max_pool2d_cuda_forward_optimized(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const int batch_size = input.size(0);
    const int channels = input.size(1);
    const int input_height = input.size(2);
    const int input_width = input.size(3);
    
    // Compute output dimensions
    const int output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const int output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int total_elements = batch_size * channels * output_height * output_width;
    const int threads = 256;  // Increased thread count for better performance
    const int blocks = (total_elements + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool2d_cuda_forward_optimized", ([&] {
        max_pool2d_kernel_optimized<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward_optimized, "Max Pool 2D forward with optimized load balancing (CUDA)");
}
