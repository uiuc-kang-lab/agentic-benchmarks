#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__device__ __forceinline__ float4 tanh_vec4(float4 val) {
    float4 result;
    result.x = tanhf(val.x);
    result.y = tanhf(val.y);
    result.z = tanhf(val.z);
    result.w = tanhf(val.w);
    return result;
}

template <typename scalar_t>
__global__ void tanh_kernel_vectorized(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int vec4_size = size / 4;
    
    // Process 4 elements at a time using float4
    const float4* input4 = reinterpret_cast<const float4*>(input);
    float4* output4 = reinterpret_cast<float4*>(output);
    
    for (int i = idx; i < vec4_size; i += stride) {
        float4 in4 = input4[i];
        output4[i] = tanh_vec4<scalar_t>(in4);
    }
    
    // Handle remaining elements
    const int remaining_start = vec4_size * 4;
    for (int i = remaining_start + idx; i < size; i += stride) {
        output[i] = tanhf(input[i]);
    }
}

torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    
    const int threads = 256;
    const int blocks = (input.numel() / 4 + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "tanh_kernel_vectorized", ([&] {
        tanh_kernel_vectorized<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            input.numel()
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Tanh forward vectorized (CUDA)");
}