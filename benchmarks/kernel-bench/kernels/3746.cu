#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void softplus_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < size) {
        // Softplus formula: f(x) = log(1 + exp(x))
        // Using numerically stable version to prevent overflow
        const scalar_t x = input[idx];
        if (x > 20.0) {
            // For large x, softplus(x) ≈ x to avoid overflow
            output[idx] = x;
        } else if (x < -20.0) {
            // For very negative x, softplus(x) ≈ exp(x) to avoid underflow
            output[idx] = exp(x);
        } else {
            output[idx] = log1p(exp(x));
        }
    }
}

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    const int threads = 256;
    const int blocks = (size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "softplus_forward_cuda", ([&] {
        softplus_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}