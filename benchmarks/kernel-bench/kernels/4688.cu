#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 512  // Experiment with values: 32, 64, 128, 256, 512

template <typename scalar_t>
__global__ void rms_norm_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int num_features,
    const int numel_per_batch,
    const float eps
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int total = batch_size * numel_per_batch;
    if (tid >= total) return;

    const int batch_id = tid / numel_per_batch;
    const int offset_in_batch = tid % numel_per_batch;
    const int batch_offset = batch_id * num_features * numel_per_batch;

    // Calculate sum of squares
    scalar_t sumsq = 0;
    for (int feat = 0; feat < num_features; feat++) {
        scalar_t val = input[batch_offset + feat * numel_per_batch + offset_in_batch];
        sumsq += val * val;
    }

    // Compute RMS normalization value
    scalar_t rms = sqrt(sumsq / num_features + eps);

    // Normalize features
    for (int feat = 0; feat < num_features; feat++) {
        int idx = batch_offset + feat * numel_per_batch + offset_in_batch;
        output[idx] = input[idx] / rms;
    }
}

torch::Tensor rms_norm_cuda_forward(torch::Tensor input, float eps) {
    auto output = torch::empty_like(input);

    const int batch_size = input.size(0);
    const int num_features = input.size(1);

    int numel_per_batch = 1;
    for (int i = 2; i < input.dim(); i++) {
        numel_per_batch *= input.size(i);
    }

    const int total_threads = batch_size * numel_per_batch;
    const int blocks = (total_threads + BLOCK_SIZE - 1) / BLOCK_SIZE;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "rms_norm_cuda", ([&] {
        rms_norm_kernel<scalar_t><<<blocks, BLOCK_SIZE>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            num_features,
            numel_per_batch,
            eps
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &rms_norm_cuda_forward, "RMS normalization forward (CUDA)");
}
