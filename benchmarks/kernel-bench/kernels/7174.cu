#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// This CUDA kernel performs 2D convolution on asymmetric input with a square kernel.
// It optimizes thread and block distribution to ensure even workload across threads.
// It also includes a fallback to native PyTorch conv2d for unsupported cases.

__global__ void conv2d_optimized_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int out_height,
    int out_width,
    int kernel_size,
    int stride,
    int padding,
    int dilation) {

    int n = blockIdx.x;       // batch index
    int oc = blockIdx.y;      // output channel index
    int out_y = blockIdx.z * blockDim.y + threadIdx.y;
    int out_x = threadIdx.x;

    if (out_y >= out_height || out_x >= out_width) return;

    float sum = 0.0f;

    for (int ic = 0; ic < in_channels; ++ic) {
        #pragma unroll
        for (int ky = 0; ky < kernel_size; ++ky) {
            #pragma unroll
            for (int kx = 0; kx < kernel_size; ++kx) {
                int in_y = out_y * stride - padding + ky * dilation;
                int in_x = out_x * stride - padding + kx * dilation;

                if (in_y >= 0 && in_y < in_height && in_x >= 0 && in_x < in_width) {
                    int input_idx = n * in_channels * in_height * in_width
                                  + ic * in_height * in_width
                                  + in_y * in_width
                                  + in_x;
                    int weight_idx = oc * in_channels * kernel_size * kernel_size
                                   + ic * kernel_size * kernel_size
                                   + ky * kernel_size
                                   + kx;
                    sum += input[input_idx] * weight[weight_idx];
                }
            }
        }
    }

    if (bias) {
        sum += bias[oc];
    }
    
    int output_idx = n * out_channels * out_height * out_width
                   + oc * out_height * out_width
                   + out_y * out_width
                   + out_x;
    output[output_idx] = sum;
}

// Host function that prepares the tensors and launches the CUDA kernel
// Falls back to native PyTorch conv2d for unsupported cases

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    // Check if groups is supported
    if (groups != 1) {
        if (bias.has_value()) {
            return torch::conv2d(x, weight, bias.value(), {stride, stride}, {padding, padding}, {dilation, dilation}, groups);
        } else {
            return torch::conv2d(x, weight, torch::Tensor(), {stride, stride}, {padding, padding}, {dilation, dilation}, groups);
        }
    }

    // Extract dimensions
    const int batch = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);

    const int out_channels = weight.size(0);
    const int kernel_size = weight.size(2);
    
    const int out_height = (in_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;
    const int out_width  = (in_width  + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;

    auto output = torch::zeros({batch, out_channels, out_height, out_width}, x.options());
    
    dim3 threads(out_width, 1);
    dim3 blocks(batch, out_channels, (out_height + threads.y - 1) / threads.y);

    const float* input_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = (bias.has_value()) ? bias.value().data_ptr<float>() : nullptr;
    float* output_ptr = output.data_ptr<float>();

    conv2d_optimized_kernel<<<blocks, threads>>>(
        input_ptr, weight_ptr, bias_ptr, output_ptr,
        batch, in_channels, out_channels,
        in_height, in_width, out_height, out_width,
        kernel_size, stride, padding, dilation);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Efficient CUDA forward function for 2D convolution with optimized and fallback support");
}