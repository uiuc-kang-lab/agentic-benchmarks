#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Grid-stride loop kernel for LeakyReLU
__global__ void leaky_relu_stride_kernel(const float* __restrict__ x, float* __restrict__ out, float negative_slope, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (; idx < n; idx += stride) {
        float val = x[idx];
        out[idx] = (val > 0.0f) ? val : val * negative_slope;
    }
}

// Forward function launching the grid-stride loop kernel
torch::Tensor leaky_relu_forward_stride(torch::Tensor x, float negative_slope) {
    CHECK_INPUT(x);
    
    auto out = torch::empty_like(x);
    int n = x.numel();

    // Using a block size that balances occupancy and resource usage
    const int threads = 1024;
    // Launch enough blocks to cover the workload; grid-stride loop ensures full coverage
    const int blocks = (n + threads - 1) / threads;

    leaky_relu_stride_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(), out.data_ptr<float>(), negative_slope, n
    );

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &leaky_relu_forward_stride, "LeakyReLU forward with grid-stride loop (CUDA)");
}
