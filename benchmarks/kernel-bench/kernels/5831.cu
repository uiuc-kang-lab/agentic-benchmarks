#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>

// Define an inline device function for minimum of two ints
__device__ inline int dmin(int a, int b) { return a < b ? a : b; }

// Optimized CUDA kernel for 3D max pooling that minimizes warp divergence
template <typename scalar_t>
__global__ void max_pool3d_forward_optimized_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t* __restrict__ indices,
    const int batch_size,
    const int channels,
    const int input_d, const int input_h, const int input_w,
    const int output_d, const int output_h, const int output_w,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation) {

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total = batch_size * channels * output_d * output_h * output_w;
    if (idx >= total) return;

    // Compute output indices
    const int w_out = idx % output_w;
    const int h_out = (idx / output_w) % output_h;
    const int d_out = (idx / (output_w * output_h)) % output_d;
    const int c = (idx / (output_w * output_h * output_d)) % channels;
    const int b = idx / (output_w * output_h * output_d * channels);

    // Compute start positions in the input
    const int d_start = d_out * stride - padding;
    const int h_start = h_out * stride - padding;
    const int w_start = w_out * stride - padding;

    // Precompute valid loop bounds for each dimension to avoid conditional branching inside loops
    // For depth dimension
    int k_d_min = (d_start < 0) ? ((-d_start + dilation - 1) / dilation) : 0;
    int valid_d_max = (input_d - d_start + dilation - 1) / dilation;
    int k_d_max = dmin(kernel_size, valid_d_max);

    // For height dimension
    int k_h_min = (h_start < 0) ? ((-h_start + dilation - 1) / dilation) : 0;
    int valid_h_max = (input_h - h_start + dilation - 1) / dilation;
    int k_h_max = dmin(kernel_size, valid_h_max);

    // For width dimension
    int k_w_min = (w_start < 0) ? ((-w_start + dilation - 1) / dilation) : 0;
    int valid_w_max = (input_w - w_start + dilation - 1) / dilation;
    int k_w_max = dmin(kernel_size, valid_w_max);

    // Initialize max value
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    int max_index = -1;

    // Loop over the valid pooling window without further conditional checks
    for (int k_d = k_d_min; k_d < k_d_max; k_d++) {
        const int d_in = d_start + k_d * dilation;
        for (int k_h = k_h_min; k_h < k_h_max; k_h++) {
            const int h_in = h_start + k_h * dilation;
            for (int k_w = k_w_min; k_w < k_w_max; k_w++) {
                const int w_in = w_start + k_w * dilation;
                const int input_idx = (((b * channels + c) * input_d + d_in) * input_h + h_in) * input_w + w_in;
                const scalar_t val = input[input_idx];
                if (val > max_val) {
                    max_val = val;
                    max_index = input_idx;
                }
            }
        }
    }

    output[idx] = max_val;
    if (indices != nullptr) {
        indices[idx] = max_index;
    }
}

// Host function that sets up the CUDA kernel launch
torch::Tensor max_pool3d_cuda_forward_optimized(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation,
    bool return_indices,
    bool ceil_mode) {

    auto input_sizes = input.sizes();
    const int batch_size = input_sizes[0];
    const int channels = input_sizes[1];
    const int input_d = input_sizes[2];
    const int input_h = input_sizes[3];
    const int input_w = input_sizes[4];

    // Calculate output dimensions
    const int output_d = ceil_mode ?
        ceil((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    const int output_h = ceil_mode ?
        ceil((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    const int output_w = ceil_mode ?
        ceil((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);

    auto output = torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options());
    auto indices = return_indices ?
        torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options().dtype(torch::kLong)) :
        torch::Tensor();

    const int threads = 256;
    const int total = batch_size * channels * output_d * output_h * output_w;
    const int blocks = (total + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool3d_forward_cuda_optimized", ([&] {
        max_pool3d_forward_optimized_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            return_indices ? indices.data_ptr<int64_t>() : nullptr,
            batch_size, channels,
            input_d, input_h, input_w,
            output_d, output_h, output_w,
            kernel_size, stride, padding, dilation);
    }));

    if (return_indices) {
        return torch::stack({output, indices}, 0);
    }
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool3d_cuda_forward_optimized, "Max Pool 3D forward optimized (CUDA)");
}
