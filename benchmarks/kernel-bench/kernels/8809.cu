#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel using 2D block configuration for better occupancy
__global__ void conv_transpose1d_kernel(
    const float* __restrict__ x,      // [N, C_in, L_in]
    const float* __restrict__ weight, // [C_in, C_out, K_w]
    const float* __restrict__ bias,   // [C_out] or nullptr
    float* __restrict__ y,            // [N, C_out, L_out]
    int N, int C_in, int C_out, int L_in, int L_out, int K_w,
    int stride, int padding, int dilation) {

    // Use 2D thread blocks: one dimension for C_out, one for L_out
    const int l_out = blockIdx.x * blockDim.x + threadIdx.x;
    const int c_out = blockIdx.y * blockDim.y + threadIdx.y;
    const int n = blockIdx.z;

    // Check if this thread should compute an output
    if (l_out < L_out && c_out < C_out && n < N) {
        
        // Initialize the accumulator with bias if available
        float value = (bias != nullptr) ? bias[c_out] : 0.0f;
        
        // Pointer offset for the current batch
        int x_batch_offset = n * C_in * L_in;
        
        // Loop over input channels
        for (int c_in = 0; c_in < C_in; ++c_in) {
            int x_channel_offset = x_batch_offset + c_in * L_in;
            int w_offset = c_in * C_out * K_w + c_out * K_w;
            
            // Loop over kernel width positions
            for (int k_w = 0; k_w < K_w; ++k_w) {
                // Compute the corresponding input index numerator
                int l_in_nom = l_out + padding - k_w * dilation;
                // Check if l_in_nom is exactly divisible by stride
                if (l_in_nom % stride == 0) {
                    int l_in = l_in_nom / stride;
                    // Check for valid input index
                    if (l_in >= 0 && l_in < L_in) {
                        value += x[x_channel_offset + l_in] * weight[w_offset + k_w];
                    }
                }
            }
        }
        
        // Write the computed value to the output tensor
        y[idx] = value;
    }
}

// Host function that wraps the CUDA kernel launch
torch::Tensor conv_transpose1d_forward(
    py::object x_obj,
    py::object weight_obj,
    py::object bias_obj = py::none(),
    int64_t stride = 1,
    int64_t padding = 0,
    int64_t dilation = 1) {
    
    torch::Tensor x = x_obj.cast<torch::Tensor>().contiguous();
    torch::Tensor weight = weight_obj.cast<torch::Tensor>().contiguous();
    
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA device");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA device");
    
    float* bias_ptr = nullptr;
    if (!bias_obj.is_none()) {
        torch::Tensor bias = bias_obj.cast<torch::Tensor>().contiguous();
        TORCH_CHECK(bias.is_cuda(), "Bias tensor must be on CUDA device");
        bias_ptr = bias.data_ptr<float>();
    }
    
    int N = x.size(0);
    int C_in = x.size(1);
    int L_in = x.size(2);
    int K_w = weight.size(2);
    int C_out = weight.size(1);
    
    // Calculate output length
    int L_out = (L_in - 1) * stride - 2 * padding + dilation * (K_w - 1) + 1;
    auto y = torch::empty({N, C_out, L_out}, x.options());
    
    int total_elements = N * C_out * L_out;
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;
    
    conv_transpose1d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        y.data_ptr<float>(),
        N, C_in, C_out, L_in, L_out, K_w,
        stride, padding, dilation);

    TORCH_CHECK(hipGetLastError() == hipSuccess, "CUDA kernel failed");
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        &conv_transpose1d_forward,
        "Conv Transpose1D forward (CUDA) with grid-stride loop",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = 1,
        py::arg("padding") = 0,
        py::arg("dilation") = 1);
}
