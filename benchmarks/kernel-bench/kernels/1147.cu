#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// CUDA kernel using shared memory and warp-level primitives for reduction
template <typename scalar_t>
__global__ void module_fn_cuda_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ output,
    int N, int M, int K, int L) {

    extern __shared__ __align__(sizeof(scalar_t)) char smem[];
    scalar_t* shared_sum = reinterpret_cast<scalar_t*>(smem);

    int n = blockIdx.z;
    int m = blockIdx.y * blockDim.y + threadIdx.y;
    int l = blockIdx.x * blockDim.x + threadIdx.x;

    if (m < M && l < L) {
        scalar_t sum = 0;
        for (int k = threadIdx.x; k < K; k += blockDim.x) {
            scalar_t a_val = A[n * M * K + m * K + k];
            scalar_t b_val = B[k * L + l];
            sum += a_val * b_val;
        }
        shared_sum[threadIdx.x] = sum;
        __syncthreads();

        // Reduce within block using shared memory
        for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
            if (threadIdx.x < stride) {
                shared_sum[threadIdx.x] += shared_sum[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Use warp shuffle for final reduction
        if (threadIdx.x < 32) {
            sum = shared_sum[threadIdx.x];
            for (int offset = 16; offset > 0; offset /= 2) {
                sum += __shfl_down_sync(0xffffffff, sum, offset);
            }
        }

        // Write result for this block
        if (threadIdx.x == 0) {
            output[n * M * L + m * L + l] = sum;
        }
    }
}

// CUDA forward function
void module_fn_cuda_forward(
    torch::Tensor A,
    torch::Tensor B,
    torch::Tensor output) {

    int N = A.size(0);
    int M = A.size(1);
    int K = A.size(2);
    int L = B.size(1);

    const int threads_x = 32;
    const int threads_y = 32;
    const dim3 threads(threads_x, threads_y);
    const dim3 blocks((L + threads_x - 1) / threads_x, (M + threads_y - 1) / threads_y, N);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(A.scalar_type(), "module_fn_cuda_forward", ([&] {
      module_fn_cuda_kernel<scalar_t><<<blocks, threads, threads_x * sizeof(scalar_t)>>>(        
          A.data_ptr<scalar_t>(),
          B.data_ptr<scalar_t>(),
          output.data_ptr<scalar_t>(),
          N, M, K, L);
    }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in module_fn_cuda_forward: %s\n", hipGetErrorString(err));
    }
}

// C++ interface
#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x)  TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x)  CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor module_fn_forward(
    torch::Tensor A,
    torch::Tensor B) {
  CHECK_INPUT(A);
  CHECK_INPUT(B);

  auto N = A.size(0);
  auto M = A.size(1);
  auto L = B.size(1);

  auto output = torch::zeros({N, M, L}, A.options());
  module_fn_cuda_forward(A, B, output);
  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &module_fn_forward, "module_fn forward (CUDA)");
}
