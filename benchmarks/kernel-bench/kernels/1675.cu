#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   int N) {
    const int warpSize = 32;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit if out of bounds
    if (row >= N || col >= N) return;
    
    // Determine if this warp works on lower triangle
    // Use warp-aligned coordinates to check
    const int warpRow = row & ~(warpSize-1);
    const int warpCol = col & ~(warpSize-1);
    
    if (warpRow >= warpCol) {
        // This warp potentially contains valid lower triangle elements
        if (row >= col) {
            // This specific thread is in lower triangle
            float sum = 0.0f;
            #pragma unroll 4
            for (int k = col; k <= row; k++) {
                sum += A[row * N + k] * B[k * N + col];
            }
            C[row * N + col] = sum;
        } else {
            C[row * N + col] = 0.0f;
        }
    } else {
        // Entire warp is in upper triangle
        C[row * N + col] = 0.0f;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Use 32x32 thread blocks to align with warp size
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((N + 31) / 32, (N + 31) / 32);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}