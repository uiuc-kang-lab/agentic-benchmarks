#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Combined and optimized sigmoid kernel
template <typename scalar_t>
__global__ void optimized_sigmoid_kernel(const scalar_t* __restrict__ input,
                                         scalar_t* __restrict__ output,
                                         const int64_t size) {
    // Calculate global thread index.
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Process multiple elements per thread for better memory coalescing
    int stride = blockDim.x * gridDim.x;
    
    for (int i = tid; i < size; i += stride) {
        float val = static_cast<float>(-input[i]);  // Use inline negative transformation
        float exp_val = expf(val);
        float r = 1.0f / (1.0f + exp_val);  // Compute sigmoid
        output[i] = static_cast<scalar_t>(r);
    }
}

torch::Tensor forward(torch::Tensor input) {
    // Allocate output tensor.
    auto output = torch::empty_like(input);
    const int64_t size = input.numel();

    // Define CUDA kernel launch configuration.
    const int threads = 256;
    const int blocks = (size + threads - 1) / threads;

    // Dispatch to the combined and optimized CUDA kernel.
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "optimized_sigmoid_kernel", [&] {
        const auto* input_data = input.data_ptr<scalar_t>();
        auto* output_data = output.data_ptr<scalar_t>();

        optimized_sigmoid_kernel<scalar_t><<<blocks, threads>>>(input_data, output_data, size);
    });

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Sigmoid forward (CUDA) optimized");
}