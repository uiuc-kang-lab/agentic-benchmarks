#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void smooth_l1_loss_kernel(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    int n_elements
) {
    const unsigned int tid = threadIdx.x;
    const unsigned int lane_id = tid % 32;
    const unsigned int warp_id = tid / 32;
    const unsigned int idx = blockIdx.x * blockDim.x + tid;
    const unsigned int stride = gridDim.x * blockDim.x;
    
    float thread_sum = 0.0f;

    // Process multiple elements per thread
    #pragma unroll 4
    for (int i = idx; i < n_elements; i += stride) {
        const float diff = predictions[i] - targets[i];
        const float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? (0.5f * diff * diff) : (abs_diff - 0.5f);
    }

    // Warp-level reduction using shuffle
    #pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
    }

    // First thread in each warp writes to global memory
    if (lane_id == 0) {
        atomicAdd(output, thread_sum / n_elements);
    }
}

torch::Tensor smooth_l1_loss_cuda(
    torch::Tensor predictions,
    torch::Tensor targets
) {
    TORCH_CHECK(
        predictions.sizes() == targets.sizes(),
        "Input tensors must have the same shape"
    );
    TORCH_CHECK(
        predictions.is_contiguous() && targets.is_contiguous(),
        "Input tensors must be contiguous"
    );
    TORCH_CHECK(
        predictions.device().is_cuda() && targets.device().is_cuda(),
        "Inputs must be CUDA tensors"
    );

    int n = predictions.numel();
    auto output = torch::zeros({1}, predictions.options());

    const int block_size = 256;  // Multiple of warp size (32)
    const int grid_size = (n + block_size - 1) / block_size;

    smooth_l1_loss_kernel<<<grid_size, block_size>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &smooth_l1_loss_cuda, "Smooth L1 Loss (CUDA)");
}