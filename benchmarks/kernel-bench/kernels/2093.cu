#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel to compute C = tril(A * B) for lower triangular matrices A and B.
__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                       const float* __restrict__ B,
                                       float* __restrict__ C,
                                       int N) {
    // Compute the row and column indices for this thread.
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        // Only compute values for the lower triangular part.
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            float sum = 0.f;
            // For lower triangular matrices, A[i,k] is nonzero only if k <= i
            // and B[k,j] is nonzero only if j <= k.
            // Hence, only indices k from col to row contribute.
            #pragma unroll
            for (int k = col; k <= row; ++k) {
                sum += A[row * N + k] * B[k * N + col];
            }
            C[row * N + col] = sum;
        }
    }
}

// C++ interface exposed to PyTorch.
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Define block and grid dimensions.
    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threads - 1) / threads, (N + threads - 1) / threads);

    // Launch the CUDA kernel.
    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    // Check for kernel launch errors.
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}