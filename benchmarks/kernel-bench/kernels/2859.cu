#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Improved sigmoid kernel with minimal synchronization
template <typename scalar_t>
__global__ void sigmoid_kernel_optimized(const scalar_t* __restrict__ input,
                                         scalar_t* __restrict__ output,
                                         const int64_t size) {
  extern __shared__ float shared_mem[];
  const int tid = threadIdx.x;
  const int i = blockIdx.x * blockDim.x + tid;

  if (i < size) {
    shared_mem[tid] = static_cast<float>(-input[i]);
  }

  // Synchronize to ensure shared memory is fully populated
  __syncthreads();

  if (i < size) {
    float exp_val = expf(shared_mem[tid]);
    float r = 1.0f / (1.0f + exp_val);
    output[i] = static_cast<scalar_t>(r);
  }
}

torch::Tensor forward(torch::Tensor input) {
  // Allocate output tensor.
  auto output = torch::empty_like(input);
  const int64_t size = input.numel();

  // Define CUDA kernel launch configuration.
  const int threads = 256;
  const int blocks = (size + threads - 1) / threads;
  const int shared_mem_size = threads * sizeof(float);

  // Dispatch to our CUDA kernel.
  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sigmoid_kernel_optimized", [&] {
    const auto* input_data = input.data_ptr<scalar_t>();
    auto* output_data = output.data_ptr<scalar_t>();

    sigmoid_kernel_optimized<scalar_t><<<blocks, threads, shared_mem_size>>>(input_data, output_data, size);
  });

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "Optimized Sigmoid forward (CUDA)");
}