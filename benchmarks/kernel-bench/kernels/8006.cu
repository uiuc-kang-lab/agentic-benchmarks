#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor forward(
    const torch::Tensor& x,
    const torch::Tensor& weight,
    const c10::optional<torch::Tensor>& bias,
    int64_t stride,
    int64_t padding,
    int64_t output_padding,
    int64_t groups) {
    
    CHECK_INPUT(x);
    CHECK_INPUT(weight);

    // Handle optional bias tensor
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
        return torch::conv_transpose1d(
            x,
            weight,
            bias.value(),
            stride,
            padding,
            output_padding,
            groups
        );
    } else {
        return torch::conv_transpose1d(
            x,
            weight,
            torch::Tensor(), // Empty tensor for no bias
            stride,
            padding,
            output_padding,
            groups
        );
    }
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Transposed 1D convolution forward (CUDA)");
}