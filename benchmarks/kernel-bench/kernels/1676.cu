#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel that minimizes warp divergence by eliminating conditional branching
// for lower triangular evaluation. All threads run the same loop, and the result
// is masked by an indicator to ensure correctness.
__global__ void warp_uniform_tril_kernel(const float* __restrict__ A,
                                          const float* __restrict__ B,
                                          float* __restrict__ C,
                                          int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        // The loop from 'col' to 'row' naturally results
        // in zero iterations if row < col, avoiding an explicit branch
        for (int k = col; k <= row; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        // Multiply with the indicator (1 if row >= col, 0 otherwise) to ensure correctness
        int valid = (row >= col) ? 1 : 0;
        C[row * N + col] = sum * valid;
    }
}

// C++ interface exposed to PyTorch.
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Define block and grid dimensions
    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threads - 1) / threads, (N + threads - 1) / threads);

    // Launch the kernel
    warp_uniform_tril_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Warp uniform lower triangular matrix multiplication (CUDA)");
}
