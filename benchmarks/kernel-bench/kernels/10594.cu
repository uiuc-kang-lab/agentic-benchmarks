#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void cumprod_kernel_optimized(
    scalar_t* output,
    const scalar_t* input,
    const int64_t numel,
    const int64_t dim_size,
    const int64_t stride) {

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numel / dim_size) return;
    const int batch_idx = idx / stride;
    const int in_idx = idx % stride;
    
    if (idx < numel / dim_size) {
        scalar_t product = 1;
        int start_idx = batch_idx * stride * dim_size + in_idx;

        for (int i = 0; i < dim_size; i++) {
            int curr_idx = start_idx + i * stride;
            product *= input[curr_idx];
            output[curr_idx] = product;
        }
    }
}

torch::Tensor cumprod_cuda_forward_optimized(torch::Tensor input, int64_t dim) {
    auto output = torch::empty_like(input);
    
    auto sizes = input.sizes();
    auto strides = input.strides();
    
    int64_t dim_size = sizes[dim];
    int64_t stride = strides[dim];
    int64_t numel = input.numel();
    int64_t total_threads = numel / dim_size;
    
    const int threads = 256;
    const int blocks = (total_threads + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "cumprod_cuda_optimized", ([&] {
        cumprod_kernel_optimized<scalar_t><<<blocks, threads>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            numel,
            dim_size,
            stride
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &cumprod_cuda_forward_optimized, "Cumulative product forward optimized (CUDA)");
}