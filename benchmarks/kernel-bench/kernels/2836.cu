#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <type_traits>

// Device function for exponentiation preserving precision
template <typename T>
__device__ inline T myExp(T x);

template <>
__device__ inline float myExp<float>(float x) {
    return expf(x);
}

template <>
__device__ inline double myExp<double>(double x) {
    return exp(x);
}

// Union to facilitate vectorized load and store operations.
// VecT: vector type (e.g., float4 or double2); VecSize: number of scalar elements
// This ensures 128-bit aligned accesses.

template <typename scalar_t, typename VecT, int VecSize>
union VecUnion {
  VecT vec;
  scalar_t arr[VecSize];
};

// Vectorized kernel using grid-stride loop. It reads global memory via __ldg() for
// read-only accesses and assumes 128-bit alignment for loads/stores.

template <typename scalar_t, typename VecT, int VecSize>
__global__ void vectorized_sigmoid_kernel(const scalar_t* __restrict__ input,
                                           scalar_t* __restrict__ output,
                                           int64_t vec_count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < vec_count; i += stride) {
        VecUnion<scalar_t, VecT, VecSize> data;
        // Load using __ldg for optimized read-only access
        data.vec = __ldg(reinterpret_cast<const VecT*>(input) + i);

        #pragma unroll
        for (int j = 0; j < VecSize; j++) {
            scalar_t x = data.arr[j];
            scalar_t exp_val = myExp(-x);
            data.arr[j] = (scalar_t(1)) / (scalar_t(1) + exp_val);
        }
        reinterpret_cast<VecT*>(output)[i] = data.vec;
    }
}

// Scalar kernel to handle tail elements that are not a multiple of the vectorization factor.

template <typename scalar_t>
__global__ void scalar_sigmoid_kernel(const scalar_t* __restrict__ input,
                                        scalar_t* __restrict__ output,
                                        int64_t start,
                                        int64_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + start;
    if (idx < size) {
        scalar_t x = __ldg(&input[idx]);
        scalar_t exp_val = myExp(-x);
        output[idx] = (scalar_t(1)) / (scalar_t(1) + exp_val);
    }
}

// Forward function launches the kernels. It uses vectorized loads/stores for 128-bit aligned
// operations and falls back to a scalar kernel for any remaining tail elements.

torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int64_t size = input.numel();
    const int threads = 512; // Increased thread count for higher occupancy

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "fast_ldg_128_sigmoid", ([&] {
        const auto* input_data = input.data_ptr<scalar_t>();
        auto* output_data = output.data_ptr<scalar_t>();

        // Choose vectorization factor: float4 for float and double2 for double (128-bit load/store)
        int vecSize = std::is_same<scalar_t, float>::value ? 4 : 2;
        int64_t vec_elements = size / vecSize; // number of full vectorized groups
        int blocks = (vec_elements + threads - 1) / threads;

        if (vec_elements > 0) {
            if (std::is_same<scalar_t, float>::value) {
                vectorized_sigmoid_kernel<scalar_t, float4, 4><<<blocks, threads>>>(input_data, output_data, vec_elements);
            } else {
                vectorized_sigmoid_kernel<scalar_t, double2, 2><<<blocks, threads>>>(input_data, output_data, vec_elements);
            }
        }

        // Process tail elements that don’t fit into a full 128-bit vector
        int64_t vec_aligned_size = vec_elements * vecSize;
        int64_t tail = size - vec_aligned_size;
        if (tail > 0) {
            int tail_blocks = (tail + threads - 1) / threads;
            scalar_sigmoid_kernel<scalar_t><<<tail_blocks, threads>>>(input_data, output_data, vec_aligned_size, size);
        }
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "Fast LDG and 128-bit aligned Sigmoid forward (CUDA)");
}
