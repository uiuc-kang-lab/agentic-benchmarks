#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/util/Optional.h>

namespace py = pybind11;

// -----------------------------------------------------------------
// 1D convolution CUDA kernel with uniform control flow
// to minimize warp divergence by using branchless index handling.
// -----------------------------------------------------------------
__global__ void conv1d_forward_kernel_uniform(
    const float* __restrict__ x,
    const float* __restrict__ w,
    const float* __restrict__ bias_ptr, // can be null if no bias
    float* __restrict__ y,
    const int N,         // batch size
    const int C_in,      // input channels
    const int L_in,      // input length
    const int C_out,     // output channels
    const int K,         // kernel size
    const int stride,
    const int padding,
    const int dilation,
    const int groups,
    const int L_out      // output length
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = N * C_out * L_out;
    if (idx >= total) return;

    // Compute output coordinates: n, out_ch, out_pos
    int out_pos = idx % L_out;
    int out_ch  = (idx / L_out) % C_out;
    int n       = idx / (L_out * C_out);

    int group_size_out = C_out / groups;
    int group_size_in  = C_in / groups;
    int group_idx      = out_ch / group_size_out;

    float sum = 0.0f;
    int base_x = n * (C_in * L_in);
    int base_w = out_ch * (group_size_in * K);

    // Loop over the input channels in the current group
    for (int local_in_ch = 0; local_in_ch < group_size_in; local_in_ch++) {
        int in_ch = group_idx * group_size_in + local_in_ch;
        int base_x_channel = base_x + in_ch * L_in;
        int base_w_channel = local_in_ch * K;
        
        // Loop over the kernel size
        for (int k = 0; k < K; k++) {
            int in_pos = out_pos * stride + k * dilation - padding;
            // Compute clamped index and branchless validity mask
            int clamped = min(max(in_pos, 0), L_in - 1);
            int valid = ((unsigned)in_pos < (unsigned)L_in);  // 1 if valid, 0 otherwise
            float mask = (float) valid;
            
            // Use __ldg for read-only loads to improve caching
            float x_val = __ldg(&x[base_x_channel + clamped]);
            float w_val = __ldg(&w[base_w + base_w_channel + k]);
            
            sum += mask * x_val * w_val;
        }
    }

    // Add bias if provided (this branch is outside inner loop and thus minimal divergence)
    if (bias_ptr) {
        sum += __ldg(&bias_ptr[out_ch]);
    }

    int out_index = n * (C_out * L_out) + out_ch * L_out + out_pos;
    y[out_index] = sum;
}

// -----------------------------------------------------------------
// Implementation of conv1d forward using the uniform control flow kernel
// -----------------------------------------------------------------
at::Tensor conv1d_forward_impl_uniform(
    const at::Tensor& x,
    const at::Tensor& weight,
    c10::optional<at::Tensor> bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups
) {
    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.scalar_type() == at::kFloat, "x must be float32");
    TORCH_CHECK(weight.scalar_type() == at::kFloat, "weight must be float32");

    // x: [N, C_in, L_in]
    auto x_sizes = x.sizes();
    int64_t N    = x_sizes[0];
    int64_t C_in = x_sizes[1];
    int64_t L_in = x_sizes[2];

    // weight: [C_out, C_in/groups, K]
    auto w_sizes = weight.sizes();
    int64_t C_out = w_sizes[0];
    int64_t K     = w_sizes[2];

    // Compute output length
    int64_t L_out = (L_in + 2 * padding - dilation * (K - 1) - 1) / stride + 1;
    TORCH_CHECK(L_out > 0, "Calculated output length is non-positive.");

    auto y = torch::empty({N, C_out, L_out}, x.options().dtype(at::kFloat));

    const float* bias_ptr = nullptr;
    if (bias_opt.has_value() && bias_opt.value().defined()) {
        TORCH_CHECK(bias_opt.value().is_cuda(), "bias must be a CUDA tensor if provided");
        TORCH_CHECK(bias_opt.value().scalar_type() == at::kFloat, "bias must be float32");
        bias_ptr = bias_opt.value().data_ptr<float>();
    }

    int total_threads = N * C_out * L_out;
    int blockSize = 256;
    int gridSize = (total_threads + blockSize - 1) / blockSize;

    conv1d_forward_kernel_uniform<<<gridSize, blockSize>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        y.data_ptr<float>(),
        (int)N, (int)C_in, (int)L_in, (int)C_out, (int)K,
        (int)stride, (int)padding, (int)dilation, (int)groups, (int)L_out
    );
    
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "conv1d_forward_kernel_uniform failed: ", hipGetErrorString(err));

    return y;
}

// -----------------------------------------------------------------
// Pybind11 binding
// -----------------------------------------------------------------
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        [](at::Tensor x,
           at::Tensor weight,
           py::object bias_obj,
           int64_t stride,
           int64_t padding,
           int64_t dilation,
           int64_t groups) {
            c10::optional<at::Tensor> bias;
            if (!bias_obj.is_none()) {
                bias = bias_obj.cast<at::Tensor>();
            }
            return conv1d_forward_impl_uniform(x, weight, bias, stride, padding, dilation, groups);
        },
        "1D Convolution forward (CUDA) with uniform control flow to reduce warp divergence"
    );
}
