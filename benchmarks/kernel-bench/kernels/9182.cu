#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <pybind11/pybind11.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace py = pybind11;

__global__ void conv_transpose2d_kernel(
    const float* input,
    const float* weight,
    float* output,
    const int batch_size,
    const int in_channels,
    const int out_channels,
    const int in_height,
    const int in_width,
    const int kernel_height,
    const int kernel_width,
    const int out_height,
    const int out_width,
    const int stride_h,
    const int stride_w,
    const int pad_h,
    const int pad_w
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    const int total_elements = batch_size * out_channels * out_height * out_width;
    
    for (int idx = tid; idx < total_elements; idx += stride) {
        const int w = idx % out_width;
        const int h = (idx / out_width) % out_height;
        const int c = (idx / (out_width * out_height)) % out_channels;
        const int b = idx / (out_width * out_height * out_channels);
        
        float sum = 0.0f;
        
        for (int ic = 0; ic < in_channels; ic++) {
            for (int kh = 0; kh < kernel_height; kh++) {
                for (int kw = 0; kw < kernel_width; kw++) {
                    const int in_h = (h + pad_h - kh) / stride_h;
                    const int in_w = (w + pad_w - kw) / stride_w;
                    
                    if (in_h >= 0 && in_h < in_height && in_w >= 0 && in_w < in_width) {
                        const int input_idx = ((b * in_channels + ic) * in_height + in_h) * in_width + in_w;
                        const int weight_idx = ((c * in_channels + ic) * kernel_height + kh) * kernel_width + kw;
                        sum += input[input_idx] * weight[weight_idx];
                    }
                }
            }
        }
        output[idx] = sum;
    }
}

torch::Tensor conv_transpose2d_forward(
    torch::Tensor x,
    torch::Tensor weight,
    py::object bias_obj,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding
) {
    const auto batch_size = x.size(0);
    const auto in_channels = x.size(1);
    const auto in_height = x.size(2);
    const auto in_width = x.size(3);
    const auto out_channels = weight.size(1);
    const auto kernel_height = weight.size(2);
    const auto kernel_width = weight.size(3);
    
    const auto out_height = (in_height - 1) * stride[0] - 2 * padding[0] + kernel_height;
    const auto out_width = (in_width - 1) * stride[1] - 2 * padding[1] + kernel_width;
    
    auto output = torch::zeros({batch_size, out_channels, out_height, out_width},
                              x.options());
    
    const int threads = 256;
    const int blocks = (batch_size * out_channels * out_height * out_width + threads - 1) / threads;
    
    conv_transpose2d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        out_channels,
        in_height,
        in_width,
        kernel_height,
        kernel_width,
        out_height,
        out_width,
        stride[0],
        stride[1],
        padding[0],
        padding[1]
    );
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward, "Conv Transpose 2D forward",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride"),
          py::arg("padding"));
}