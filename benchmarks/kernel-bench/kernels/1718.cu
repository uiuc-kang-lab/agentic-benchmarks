#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
#define WARP_SIZE 32

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   int N) {
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    float sum = 0.0f;
    
    if (row < N && col < N) {
        if (row >= col) {  // Only compute for lower triangular part
            // Loop over blocks
            for (int b = col; b <= row; b += BLOCK_SIZE) {
                int remaining = min(BLOCK_SIZE, row - b + 1);
                
                // Load data into shared memory
                if (b + tx < N && ty < remaining) {
                    As[ty][tx] = A[row * N + (b + tx)];
                    Bs[ty][tx] = B[(b + ty) * N + col];
                } else {
                    As[ty][tx] = 0.0f;
                    Bs[ty][tx] = 0.0f;
                }
                
                __syncthreads();
                
                // Compute partial dot product
                #pragma unroll
                for (int k = 0; k < remaining; k++) {
                    sum += As[ty][k] * Bs[k][tx];
                }
                
                __syncthreads();
            }
            
            // Warp-level reduction for final sum
            #pragma unroll
            for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
                sum += __shfl_down_sync(0xffffffff, sum, offset);
            }
            
            // Write result
            if (threadIdx.x == 0) {
                C[row * N + col] = sum;
            }
        } else {
            C[row * N + col] = 0.0f;  // Upper triangular part is zero
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
                   (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}