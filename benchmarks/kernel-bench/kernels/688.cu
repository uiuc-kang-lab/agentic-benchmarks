#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

// CUDA kernel for matrix multiplication using shared memory tiling with minimized warp divergence
template <typename scalar_t>
__global__ void matmul_warp_divergence_optimized(const scalar_t* __restrict__ A, const scalar_t* __restrict__ B,
                                   scalar_t* __restrict__ C, int M, int K, int N) {
    __shared__ scalar_t sA[TILE_WIDTH][TILE_WIDTH];
    __shared__ scalar_t sB[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y; // M dimension
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x; // N dimension

    scalar_t value = 0;

    // Loop over tiles
    for (int t = 0; t < (K + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {
        // Load elements into shared memory
        int tiledCol = t * TILE_WIDTH + threadIdx.x;
        int tiledRow = t * TILE_WIDTH + threadIdx.y;

        sA[threadIdx.y][threadIdx.x] = (row < M && tiledCol < K) ? A[row * K + tiledCol] : 0;
        sB[threadIdx.y][threadIdx.x] = (tiledRow < K && col < N) ? B[tiledRow * N + col] : 0;

        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i) {
            value += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Write to output
    if (row < M && col < N) {
        C[row * N + col] = value;
    }
}

// Forward function
torch::Tensor module_fn(torch::Tensor A, torch::Tensor B) {
    // Ensure input tensors are CUDA tensors
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input tensor B must be a CUDA tensor");

    // Get matrix dimensions
    int64_t M = A.size(0);
    int64_t K = A.size(1);
    int64_t N = B.size(1);

    // Check dimensions compatibility
    TORCH_CHECK(K == B.size(0), "Inner dimensions of A and B must match");

    // Allocate output tensor
    auto C = torch::empty({M, N}, A.options());

    // Define block and grid dimensions
    dim3 threads_per_block(TILE_WIDTH, TILE_WIDTH);
    dim3 num_blocks((N + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);

    // Launch the CUDA kernel
    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matmul_warp_divergence_optimized", ([&] {
        matmul_warp_divergence_optimized<scalar_t><<<num_blocks, threads_per_block>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, K, N);
    }));

    // Wait for all kernels to finish
    hipDeviceSynchronize();

    return C;
}

// Binding code
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &module_fn, "Matrix multiplication forward (CUDA)");
}
