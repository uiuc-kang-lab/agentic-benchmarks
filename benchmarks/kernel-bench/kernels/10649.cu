#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Device function to compute cumulative product for a single sequence
// This function is modular and can be reused for different data types
// and configurations

template <typename scalar_t>
__device__ void compute_cumprod_stride(
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ input,
    const int64_t base_offset,
    const int64_t dim_size,
    const int64_t stride) {
    // Use shared memory for intermediate products to reduce global memory accesses
    __shared__ scalar_t shared_products[32];  // One per warp
    const int warp_id = threadIdx.x / 32;
    scalar_t product = 1;
    
    // Prefetch data to registers to reduce memory latency
    scalar_t curr_val;
    for (int i = 0; i < dim_size; i++) {
        const int64_t offset = base_offset + i * stride;
        curr_val = input[offset];
        product *= curr_val;
        
        // Store intermediate results in shared memory periodically
        if ((i + 1) % 32 == 0) {
            shared_products[warp_id] = product;
            __syncwarp();
        }
        
        output[offset] = product;
    }
}

// Kernel function that utilizes the device function for cumulative product
// Utilizing stride loop to handle workloads larger than available threads

template <typename scalar_t>
__global__ void cumprod_kernel_stride(
    scalar_t* output,
    const scalar_t* input,
    const int64_t numel,
    const int64_t dim_size,
    const int64_t stride) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int grid_size = gridDim.x * blockDim.x;
    for (int current_idx = idx; current_idx < numel / dim_size; current_idx += grid_size) {
        const int batch_idx = current_idx / stride;
        const int in_idx = current_idx % stride;
        const int64_t base_offset = batch_idx * (stride * dim_size) + in_idx;
        compute_cumprod_stride(output, input, base_offset, dim_size, stride);
    }
}

torch::Tensor cumprod_cuda_forward_stride(torch::Tensor input, int64_t dim) {
    auto output = torch::empty_like(input);
    
    // Get tensor properties
    auto sizes = input.sizes();
    auto strides = input.strides();
    
    // Calculate dimension properties
    int64_t dim_size = sizes[dim];
    int64_t stride = strides[dim];
    int64_t numel = input.numel();
    
    // Calculate total number of elements to process
    int64_t total_threads = numel / dim_size;
    
    // CUDA kernel launch parameters
    const int threads = 512;
    const int blocks = (total_threads + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "cumprod_cuda_stride", ([&] {
        cumprod_kernel_stride<scalar_t><<<blocks, threads>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            numel,
            dim_size,
            stride
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &cumprod_cuda_forward_stride, "Cumulative product forward with stride loop (CUDA)");
}