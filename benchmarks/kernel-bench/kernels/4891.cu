#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

// Optimized warp-level reduction using shuffle intrinsics
__inline__ __device__ float warpReduceSum(float val) {
    #pragma unroll
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__global__ void l1_norm_hybrid_kernel(const float* __restrict__ x,
                                    float* __restrict__ out,
                                    const int N,
                                    const int D) {
    extern __shared__ float sdata[];
    const int row = blockIdx.x;
    const int tid = threadIdx.x;
    const int step = blockDim.x;
    const int lane = tid & (warpSize-1);
    const int wid = tid / warpSize;

    // Step 1: Compute local sum with vectorized loads when possible
    float sum = 0.0f;
    if (D >= 4) {
        const float4* x4 = reinterpret_cast<const float4*>(x + row * D);
        for (int col = tid; col < D/4; col += step) {
            float4 vals = x4[col];
            sum += fabsf(vals.x) + fabsf(vals.y) + fabsf(vals.z) + fabsf(vals.w);
        }
        // Handle remaining elements
        for (int col = tid + (D/4)*4; col < D; col += step) {
            sum += fabsf(x[row * D + col]);
        }
    } else {
        for (int col = tid; col < D; col += step) {
            sum += fabsf(x[row * D + col]);
        }
    }

    // Step 2: Warp-level reduction
    sum = warpReduceSum(sum);

    // Step 3: Store warp results to shared memory
    if (lane == 0) {
        sdata[wid] = sum;
    }
    __syncthreads();

    // Step 4: Final reduction with first warp
    if (tid < 32) {
        float warp_sum = (tid < (step/warpSize)) ? sdata[tid] : 0.0f;
        warp_sum = warpReduceSum(warp_sum);
        
        if (tid == 0) {
            warp_sum = (warp_sum == 0.0f) ? 1e-12f : warp_sum;
            sdata[0] = warp_sum;
        }
    }
    __syncthreads();
    
    const float total = sdata[0];

    // Step 5: Normalize with vectorized stores when possible
    if (D >= 4) {
        float4* out4 = reinterpret_cast<float4*>(out + row * D);
        const float4* x4 = reinterpret_cast<const float4*>(x + row * D);
        for (int col = tid; col < D/4; col += step) {
            float4 vals = x4[col];
            vals.x /= total;
            vals.y /= total;
            vals.z /= total;
            vals.w /= total;
            out4[col] = vals;
        }
        // Handle remaining elements
        for (int col = tid + (D/4)*4; col < D; col += step) {
            out[row * D + col] = x[row * D + col] / total;
        }
    } else {
        for (int col = tid; col < D; col += step) {
            out[row * D + col] = x[row * D + col] / total;
        }
    }
}

torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA.");
    TORCH_CHECK(x.dim() == 2, "Expected 2D tensor.");
    x = x.contiguous();

    auto out = torch::empty_like(x);
    const int N = x.size(0);
    const int D = x.size(1);
    const int threads = std::min<int>(1024, ((D + 3)/4) * 4); // Align with vector loads
    const int shared_mem_size = (threads/32) * sizeof(float);

    l1_norm_hybrid_kernel<<<N, threads, shared_mem_size>>>(
        x.data_ptr<float>(),
        out.data_ptr<float>(),
        N, D
    );

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "L1 Normalization forward pass (CUDA with hybrid optimizations)");
}