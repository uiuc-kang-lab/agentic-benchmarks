#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

// CUDA kernel for matrix multiplication with minimized warp divergence
// The kernel distinguishes between interior blocks (fully within matrix bounds) and boundary blocks.
// For interior blocks, bounds checks are omitted to ensure uniform control flow and avoid divergent branching.

template <typename scalar_t>
__global__ void matmul_cuda_kernel(const scalar_t* __restrict__ A,
                                     const scalar_t* __restrict__ B,
                                     scalar_t* __restrict__ C,
                                     int M, int K, int N) {
    // Compute global row and column indices
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    scalar_t value = 0;

    // Determine if this block is completely interior (i.e. no boundary checks needed).
    // If (blockIdx.x+1)*TILE_WIDTH <= N and (blockIdx.y+1)*TILE_WIDTH <= M, the entire tile is within bounds.
    bool interior_block = ((blockIdx.x + 1) * TILE_WIDTH <= N) && ((blockIdx.y + 1) * TILE_WIDTH <= M);

    __shared__ scalar_t sA[TILE_WIDTH][TILE_WIDTH];
    __shared__ scalar_t sB[TILE_WIDTH][TILE_WIDTH];

    int num_tiles = (K + TILE_WIDTH - 1) / TILE_WIDTH;
    for (int t = 0; t < num_tiles; ++t) {
        int tiledA_col = t * TILE_WIDTH + threadIdx.x;
        int tiledB_row = t * TILE_WIDTH + threadIdx.y;

        // For interior blocks, all accesses are within bounds so we avoid conditional checks.
        if (interior_block) {
            sA[threadIdx.y][threadIdx.x] = __ldg(&A[row * K + tiledA_col]);
            sB[threadIdx.y][threadIdx.x] = __ldg(&B[tiledB_row * N + col]);
        } else {
            // For boundary blocks, use a ternary operator for a branchless-like selection
            sA[threadIdx.y][threadIdx.x] = (row < M && tiledA_col < K) ? __ldg(&A[row * K + tiledA_col]) : static_cast<scalar_t>(0);
            sB[threadIdx.y][threadIdx.x] = (col < N && tiledB_row < K) ? __ldg(&B[tiledB_row * N + col]) : static_cast<scalar_t>(0);
        }

        __syncthreads();

        #pragma unroll
        for (int i = 0; i < TILE_WIDTH; ++i) {
            value += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }
        __syncthreads();
    }

    // Write the computed result if within output bounds
    if (row < M && col < N) {
        C[row * N + col] = value;
    }
}

// Forward function called from Python
torch::Tensor module_fn(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input tensor B must be a CUDA tensor");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);
    TORCH_CHECK(K == B.size(0), "Inner dimensions of A and B must match");

    auto C = torch::empty({M, N}, A.options());

    dim3 threads(TILE_WIDTH, TILE_WIDTH);
    dim3 blocks((N + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matmul_cuda_kernel", ([&] {
        matmul_cuda_kernel<scalar_t><<<blocks, threads>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, K, N);
    }));

    hipDeviceSynchronize();
    return C;
}

// Pybind11 module binding
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &module_fn, "Matrix multiplication with minimized warp divergence");
}
