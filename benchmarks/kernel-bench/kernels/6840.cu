#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>

// Optimized CUDA kernel using 2D thread mapping
// Each thread now directly computes for an (outer, inner) pair
__global__ void argmax_kernel_2d(
    const float* __restrict__ x,
    int64_t* __restrict__ indices,
    const int outerSize,
    const int dimSize,
    const int innerSize) {
    // Map the two dimensions: blockIdx.y*blockDim.y + threadIdx.y -> outer index, and 
    // blockIdx.x*blockDim.x + threadIdx.x -> inner index
    int outer_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int inner_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (outer_idx < outerSize && inner_idx < innerSize) {
        int start_offset = outer_idx * (dimSize * innerSize) + inner_idx;
        float max_val = x[start_offset];
        int max_idx = 0;

        // Iterate along the dimension to compute argmax
        for (int d = 1; d < dimSize; d++) {
            float val = x[start_offset + d * innerSize];
            if (val > max_val) {
                max_val = val;
                max_idx = d;
            }
        }

        // Store the computed index
        indices[outer_idx * innerSize + inner_idx] = max_idx;
    }
}

// Host function to launch the CUDA kernel with 2D grid configuration
torch::Tensor argmax_forward_cuda(const torch::Tensor& x, const int64_t dim) {
    TORCH_CHECK(x.scalar_type() == at::kFloat, "Only float32 is supported.");
    auto x_contig = x.contiguous();

    auto sizes = x_contig.sizes();
    int ndim = x_contig.dim();
    TORCH_CHECK(dim >= 0 && dim < ndim, "Invalid dim for argmax.");

    // Compute dimensions: outerSize, dimSize, and innerSize
    int outerSize = 1;
    for (int d = 0; d < dim; d++) {
        outerSize *= sizes[d];
    }
    int dimSize = sizes[dim];
    int innerSize = 1;
    for (int d = dim + 1; d < ndim; d++) {
        innerSize *= sizes[d];
    }

    // Output tensor shape: remove the argmax dimension
    std::vector<int64_t> out_sizes;
    for (int d = 0; d < ndim; d++) {
        if (d == dim) continue;
        out_sizes.push_back(sizes[d]);
    }
    auto options = torch::TensorOptions()
                       .device(x.device())
                       .dtype(torch::kLong);
    auto indices = torch::empty(out_sizes, options);

    // Configure block dimensions optimally for 2D mapping
    dim3 block(32, 8);
    dim3 grid((innerSize + block.x - 1) / block.x,
              (outerSize + block.y - 1) / block.y);

    argmax_kernel_2d<<<grid, block>>>(
        x_contig.data_ptr<float>(),
        indices.data_ptr<int64_t>(),
        outerSize,
        dimSize,
        innerSize
    );

    return indices;
}

// Pybind11 binding
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &argmax_forward_cuda, "ArgMax CUDA forward with 2D thread mapping");
}
