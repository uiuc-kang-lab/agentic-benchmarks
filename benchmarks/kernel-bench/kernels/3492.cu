#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// GELU device function for float and double without shared memory usage
template <typename scalar_t>
__device__ inline scalar_t gelu_function(scalar_t x);

template <>
__device__ inline float gelu_function<float>(float x) {
    // GELU(x) = x * 0.5f * (1 + erf(x / sqrt(2)))
    return x * 0.5f * (1.0f + erff(x / 1.4142135623730951f));
}

template <>
__device__ inline double gelu_function<double>(double x) {
    return x * 0.5 * (1.0 + erf(x / 1.4142135623730951));
}

// Kernel that computes GELU elementwise using a grid-stride loop
// No shared memory is used, so no __syncthreads() are necessary, thus minimizing synchronization overhead.
template <typename scalar_t>
__global__ void gelu_direct_kernel(const scalar_t* __restrict__ input,
                                    scalar_t* __restrict__ output,
                                    size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = idx; i < n; i += stride) {
        const scalar_t val = __ldg(&input[i]);
        output[i] = gelu_function<scalar_t>(val);
    }
}

// Forward function callable from Python
torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
    auto output = torch::empty_like(x);
    size_t numel = x.numel();

    // Use 256 threads per block
    const int threads = 256;
    const int blocks = (numel + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "gelu_direct_cuda", ([&] {
        gelu_direct_kernel<scalar_t><<<blocks, threads>>>(
            x.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            numel
        );
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "GELU activation forward (CUDA, no excessive synchronization)");
}
