#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

namespace py = pybind11;

// This kernel implements conv_transpose2d using a gather approach with shared memory optimization.
// Each thread computes one output pixel by gathering contributions from all input channels and the kernel window.
// Shared memory is used to cache input data to reduce global memory access latency.

__global__ void conv_transpose2d_forward_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int N,
    int in_channels,
    int in_h,
    int in_w,
    int out_channels,
    int kernel_h,
    int kernel_w,
    int out_h,
    int out_w,
    int stride_h,
    int stride_w,
    int pad_h,
    int pad_w,
    bool has_bias
) {
    extern __shared__ float shared_input[];

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = N * out_channels * out_h * out_w;
    if (index >= total) return;

    // Decode the output index into (n, oc, out_y, out_x) assuming row-major layout
    int out_x = index % out_w;
    int tmp = index / out_w;
    int out_y = tmp % out_h;
    tmp = tmp / out_h;
    int oc = tmp % out_channels;
    int n = tmp / out_channels;

    float sum = has_bias ? bias[oc] : 0.0f;

    // Load input data into shared memory
    int input_channel_stride = in_h * in_w;
    int weight_kernel_stride = kernel_h * kernel_w;
    for (int ic = 0; ic < in_channels; ic++) {
        int input_offset = (n * in_channels + ic) * input_channel_stride;
        for (int i = threadIdx.x; i < input_channel_stride; i += blockDim.x) {
            shared_input[i] = input[input_offset + i];
        }
        __syncthreads();

        // Optimized loop ordering: iterate over kernel spatial dimensions first to minimize redundant computations
        int base_y = out_y + pad_h;
        int base_x = out_x + pad_w;
        for (int ky = 0; ky < kernel_h; ky++) {
            int t_y = base_y - ky;
            if (t_y % stride_h != 0) continue;  // Skip if not aligned
            int in_y = t_y / stride_h;
            if (in_y < 0 || in_y >= in_h) continue;
            for (int kx = 0; kx < kernel_w; kx++) {
                int t_x = base_x - kx;
                if (t_x % stride_w != 0) continue;  // Skip if not aligned
                int in_x = t_x / stride_w;
                if (in_x < 0 || in_x >= in_w) continue;
                // Precompute offsets for input and weight for this kernel location
                int input_idx = in_y * in_w + in_x;
                int weight_idx = (ic * out_channels + oc) * weight_kernel_stride + ky * kernel_w + kx;
                sum += shared_input[input_idx] * weight[weight_idx];
            }
        }
        __syncthreads();
    }

    output[index] = sum;
}


torch::Tensor conv_transpose2d_forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding
) {
    // Input shape: [N, in_channels, in_h, in_w]
    auto N = input.size(0);
    auto in_channels = input.size(1);
    auto in_h = input.size(2);
    auto in_w = input.size(3);

    // Weight shape: [in_channels, out_channels, kernel_h, kernel_w]
    auto out_channels = weight.size(1);
    auto kernel_h = weight.size(2);
    auto kernel_w = weight.size(3);

    int stride_h = stride[0];
    int stride_w = stride[1];
    int pad_h = padding[0];
    int pad_w = padding[1];

    // Compute output dimensions based on standard transposed convolution formula
    int out_h = (in_h - 1) * stride_h - 2 * pad_h + kernel_h;
    int out_w = (in_w - 1) * stride_w - 2 * pad_w + kernel_w;

    auto output = torch::zeros({N, out_channels, out_h, out_w}, input.options());

    int total = N * out_channels * out_h * out_w;
    int threads = 256;
    int blocks = (total + threads - 1) / threads;

    bool has_bias = (bias.has_value() && bias.value().numel() > 0);
    const float* bias_ptr = has_bias ? bias.value().data_ptr<float>() : nullptr;

    size_t shared_mem_size = in_h * in_w * sizeof(float);

    conv_transpose2d_forward_kernel<<<blocks, threads, shared_mem_size, at::cuda::getCurrentCUDAStream()>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        N,
        in_channels,
        in_h,
        in_w,
        out_channels,
        kernel_h,
        kernel_w,
        out_h,
        out_w,
        stride_h,
        stride_w,
        pad_h,
        pad_w,
        has_bias
    );

    return output;
}


// Entry point from Python
torch::Tensor conv_transpose2d_forward(
    torch::Tensor input,
    torch::Tensor weight,
    py::object bias_obj,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding
) {
    c10::optional<torch::Tensor> bias = c10::nullopt;
    if (!bias_obj.is_none()) {
        bias = bias_obj.cast<torch::Tensor>();
    }
    return conv_transpose2d_forward_cuda(input, weight, bias, stride, padding);
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward, "Conv Transpose 2D forward (gather approach with shared memory)",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride"),
          py::arg("padding"));
}
