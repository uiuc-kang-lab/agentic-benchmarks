#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define BLOCK_DIM 32
#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

static hipblasHandle_t handle = nullptr;

__global__ void matmul_kernel_2d(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    const int M, const int N, const int K) {
    
    __shared__ float As[BLOCK_DIM][BLOCK_DIM];
    __shared__ float Bs[BLOCK_DIM][BLOCK_DIM];
    
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int row = blockIdx.y * BLOCK_DIM + ty;
    const int col = blockIdx.x * BLOCK_DIM + tx;
    
    float sum = 0.0f;
    
    for (int tile = 0; tile < (K + BLOCK_DIM - 1) / BLOCK_DIM; ++tile) {
        const int k_idx = tile * BLOCK_DIM;
        
        // Load A tile with proper boundary checking
        if (row < M && k_idx + tx < K) {
            As[ty][tx] = A[row * K + k_idx + tx];
        } else {
            As[ty][tx] = 0.0f;
        }
        
        // Load B tile with proper boundary checking
        if (k_idx + ty < K && col < N) {
            Bs[ty][tx] = B[(k_idx + ty) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        #pragma unroll
        for (int k = 0; k < BLOCK_DIM; ++k) {
            sum += As[ty][k] * Bs[k][tx];
        }
        
        __syncthreads();
    }
    
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

void matrix_multiply_cuda(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);

    const int M = A.size(0);
    const int K = A.size(1);
    const int N = B.size(1);

    const float* d_A = A.data_ptr<float>();
    const float* d_B = B.data_ptr<float>();
    float* d_C = C.data_ptr<float>();

    dim3 threads(BLOCK_DIM, BLOCK_DIM);
    dim3 blocks((N + BLOCK_DIM - 1) / BLOCK_DIM,
                (M + BLOCK_DIM - 1) / BLOCK_DIM);

    matmul_kernel_2d<<<blocks, threads>>>(d_A, d_B, d_C, M, N, K);
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    const int M = A.size(0);
    const int N = B.size(1);

    auto options = torch::TensorOptions()
        .dtype(A.dtype())
        .device(A.device())
        .requires_grad(false);
    
    torch::Tensor C = torch::empty({M, N}, options);
    matrix_multiply_cuda(A, B, C);
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized 2D matrix multiplication (CUDA)");
}