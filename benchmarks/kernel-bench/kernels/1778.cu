#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                     const float* __restrict__ B,
                                     float* __restrict__ C,
                                     int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            float sum = 0.f;
            int k = col;
            
            // Unroll loop by 4 for better performance
            #pragma unroll 4
            for (; k <= row - 3; k += 4) {
                sum += A[row * N + k] * B[k * N + col];
                sum += A[row * N + (k+1)] * B[(k+1) * N + col];
                sum += A[row * N + (k+2)] * B[(k+2) * N + col];
                sum += A[row * N + (k+3)] * B[(k+3) * N + col];
            }
            
            // Handle remaining elements
            for (; k <= row; k++) {
                sum += A[row * N + k] * B[k * N + col];
            }
            
            C[row * N + col] = sum;
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threads - 1) / threads, (N + threads - 1) / threads);

    // Create CUDA streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // Asynchronously copy data to device
    hipMemcpyAsync(A.data_ptr<float>(), A.data_ptr<float>(), A.numel() * sizeof(float), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(B.data_ptr<float>(), B.data_ptr<float>(), B.numel() * sizeof(float), hipMemcpyHostToDevice, stream2);

    // Launch the CUDA kernel in stream1
    triangular_mm_kernel<<<numBlocks, threadsPerBlock, 0, stream1>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    // Asynchronously copy result back to host
    hipMemcpyAsync(C.data_ptr<float>(), C.data_ptr<float>(), C.numel() * sizeof(float), hipMemcpyDeviceToHost, stream1);

    // Synchronize streams
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // Destroy streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized triangular matrix multiplication with unrolling and streams (CUDA)");
}