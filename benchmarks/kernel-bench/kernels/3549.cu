#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Device helper: define an inline exponential function for float and double.
template <typename scalar_t>
__device__ inline scalar_t my_exp(scalar_t x);

template <>
__device__ inline float my_exp<float>(float x) {
    return expf(x);
}

template <>
__device__ inline double my_exp<double>(double x) {
    return exp(x);
}

// CUDA kernel that applies the SELU activation to each element using shared memory.
template <typename scalar_t>
__global__ void selu_kernel_shared(const scalar_t* __restrict__ input,
                                   scalar_t* __restrict__ output,
                                   size_t numel) {
    __shared__ scalar_t shared_input[1024];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    
    if (idx < numel) {
        // Load data into shared memory
        shared_input[tid] = input[idx];
        __syncthreads();

        scalar_t x = shared_input[tid];
        // SELU activation: lambda * (x if x > 0 else alpha * (exp(x) - 1))
        scalar_t result = (x > static_cast<scalar_t>(0))
                              ? x
                              : static_cast<scalar_t>(1.67326324235437728481) *
                                    (my_exp(x) - static_cast<scalar_t>(1));
        output[idx] = static_cast<scalar_t>(1.05070098735548049342) * result;
    }
}

// Host function that launches the CUDA SELU kernel with shared memory.
torch::Tensor selu_forward_shared(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    auto output = torch::empty_like(input);
    const size_t numel = input.numel();
    const int threads = 1024;
    const int blocks = (numel + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "selu_forward_shared_cuda", ([&] {
        const scalar_t *input_ptr = input.data_ptr<scalar_t>();
        scalar_t *output_ptr = output.data_ptr<scalar_t>();
        selu_kernel_shared<scalar_t><<<blocks, threads, threads * sizeof(scalar_t)>>>(input_ptr, output_ptr, numel);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward_shared", &selu_forward_shared, "SELU Activation Forward with Shared Memory (CUDA)");
}