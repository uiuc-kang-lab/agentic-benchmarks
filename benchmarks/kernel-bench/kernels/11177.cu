#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// This kernel ensures memory coalescing by aligning global memory accesses.
// Threads in a warp read consecutive memory locations via vectorized float4 loads.
__global__ void smooth_l1_loss_coalesced_kernel(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    int n_elements
) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;
    int stride = blockDim.x * gridDim.x;

    float thread_sum = 0.0f;

    // Process data in groups of 4 elements using vectorized loads, which are coalesced
    int vec_count = n_elements / 4;
    const float4* pred4 = reinterpret_cast<const float4*>(predictions);
    const float4* targ4 = reinterpret_cast<const float4*>(targets);

    for (int i = idx; i < vec_count; i += stride) {
        // Each thread reads a contiguous float4 from global memory
        float4 p = __ldg(pred4 + i);
        float4 t = __ldg(targ4 + i);

        float diff = p.x - t.x;
        float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;

        diff = p.y - t.y;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;

        diff = p.z - t.z;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;

        diff = p.w - t.w;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;
    }

    // Process any remaining elements with scalar loads
    int remainderStart = vec_count * 4;
    for (int i = remainderStart + idx; i < n_elements; i += stride) {
        float diff = __ldg(predictions + i) - __ldg(targets + i);
        float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;
    }

    // Perform block-level reduction using shared memory
    __shared__ float shared_mem[256];
    shared_mem[tid] = thread_sum;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_mem[tid] += shared_mem[tid + s];
        }
        __syncthreads();
    }

    // Atomic addition of the block result to the global output (average loss)
    if (tid == 0) {
        atomicAdd(output, shared_mem[0] / n_elements);
    }
}

// Host wrapper for the CUDA kernel
torch::Tensor smooth_l1_loss_coalesced(
    torch::Tensor predictions,
    torch::Tensor targets
) {
    TORCH_CHECK(predictions.sizes() == targets.sizes(), "Input tensors must have the same shape");
    TORCH_CHECK(predictions.is_contiguous() && targets.is_contiguous(), "Input tensors must be contiguous");
    TORCH_CHECK(predictions.device().is_cuda() && targets.device().is_cuda(), "Inputs must be CUDA tensors");

    int n = predictions.numel();
    auto output = torch::zeros({1}, predictions.options());

    const int block_size = 256;
    int grid_size = (n / 4 + block_size - 1) / block_size;
    grid_size = grid_size > 0 ? grid_size : 1;

    smooth_l1_loss_coalesced_kernel<<<grid_size, block_size>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &smooth_l1_loss_coalesced, "Smooth L1 Loss with coalesced vectorized access");
}
