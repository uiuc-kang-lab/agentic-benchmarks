#include "hip/hip_runtime.h"
/*
Hybrid Matrix Multiplication Extension
This implementation combines a custom tiled CUDA kernel for small matrices and cuBLAS for larger matrices.
For small matrix sizes (e.g. <= 128x128), the custom kernel minimizes launch overhead.
For larger matrices, cuBLAS leverages highly optimized libraries and GPU tensor cores.
*/

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define TILE_SIZE 32

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Static cuBLAS handle to avoid recreation overhead
static hipblasHandle_t handle = nullptr;

// Custom tiled matrix multiplication kernel
__global__ void matmul_kernel_2d(const float* __restrict__ A,
                                 const float* __restrict__ B,
                                 float* __restrict__ C,
                                 const int M, const int N, const int K) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    // Block indices
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    // Thread indices
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    // Compute row and col for C
    const int row = by * TILE_SIZE + ty;
    const int col = bx * TILE_SIZE + tx;

    float sum = 0.0f;

    // Loop over tiles
    for (int tile = 0; tile < (K + TILE_SIZE - 1) / TILE_SIZE; ++tile) {
        // Load A tile
        if (row < M && tile * TILE_SIZE + tx < K) {
            As[ty][tx] = A[row * K + tile * TILE_SIZE + tx];
        } else {
            As[ty][tx] = 0.0f;
        }

        // Load B tile
        if (tile * TILE_SIZE + ty < K && col < N) {
            Bs[ty][tx] = B[(tile * TILE_SIZE + ty) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }

        __syncthreads();

        // Compute partial dot product using the tile
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();
    }

    // Write the result
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

// Hybrid matrix multiplication: chooses custom kernel for small matrices, cuBLAS for larger ones
void matrix_multiply_cuda(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);

    const int M = A.size(0);
    const int K = A.size(1);
    const int N = B.size(1);

    const float* d_A = A.data_ptr<float>();
    const float* d_B = B.data_ptr<float>();
    float* d_C = C.data_ptr<float>();

    // Heuristic: use custom kernel for small matrices, cuBLAS otherwise.
    if (M <= 128 && N <= 128 && K <= 128) {
        // Launch custom tiled kernel
        dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
        dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
        matmul_kernel_2d<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);
    } else {
        // Initialize cuBLAS handle if needed
        if (handle == nullptr) {
            hipblasCreate(&handle);
            // Optionally, set math mode to use Tensor Cores if available
            hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);
        }

        const float alpha = 1.0f;
        const float beta = 0.0f;

        // Note: cuBLAS assumes column-major order. Here we use arguments in a way that allows using row-major data.
        // We swap A and B pointers so that C = A*B is computed correctly.
        hipblasSgemm(handle,
                    HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N, M, K,
                    &alpha,
                    d_B, N,  // B's leading dimension
                    d_A, K,  // A's leading dimension
                    &beta,
                    d_C, N); // C's leading dimension
    }
}

// PyTorch forward interface
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    const int M = A.size(0);
    const int N = B.size(1);

    auto options = torch::TensorOptions()
                       .dtype(A.dtype())
                       .device(A.device())
                       .requires_grad(false);
    
    torch::Tensor C = torch::empty({M, N}, options);
    matrix_multiply_cuda(A, B, C);
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Hybrid matrix multiplication (CUDA): custom kernel for small matrices and cuBLAS for large matrices");
}
