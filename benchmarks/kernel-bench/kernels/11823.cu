#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ inline float compute_kldiv_value(float log_pred, float target) {
    return __expf(log_pred) - target * log_pred;
}

__device__ inline float warp_reduce_sum(float val) {
    for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xffffffff, val, offset);
    return val;
}

__device__ inline float block_reduce_sum(float val, int tid, int block_size) {
    __shared__ float warp_sums[32];
    
    int warp_id = tid / 32;
    int lane_id = tid % 32;

    val = warp_reduce_sum(val);

    if (lane_id == 0) {
        warp_sums[warp_id] = val;
    }
    __syncthreads();

    if (warp_id == 0) {
        val = (lane_id < (block_size + 31) / 32) ? warp_sums[lane_id] : 0.0f;
        val = warp_reduce_sum(val);
    }

    __syncthreads();
    return val;
}

__global__ void kl_div_kernel_combined(
    const float* log_predictions,
    const float* targets,
    float* output,
    const int n) {

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;
    int stride = blockDim.x * gridDim.x;

    float local_sum = 0.0f;
    int vec_count = n / 4;

    for (int i = gid; i < vec_count; i += stride) {
        int base = i * 4;
        float4 log_vec = reinterpret_cast<const float4*>(log_predictions)[i];
        float4 target_vec = reinterpret_cast<const float4*>(targets)[i];

        local_sum += compute_kldiv_value(log_vec.x, target_vec.x)
                   + compute_kldiv_value(log_vec.y, target_vec.y)
                   + compute_kldiv_value(log_vec.z, target_vec.z)
                   + compute_kldiv_value(log_vec.w, target_vec.w);
    }

    float block_sum = block_reduce_sum(local_sum, tid, blockDim.x);

    if (tid == 0) {
        atomicAdd(output, block_sum);
    }

    if (blockIdx.x == 0 && tid == 0) {
        float tail_sum = 0.0f;
        int tail_start = vec_count * 4;
        for (int j = tail_start; j < n; j++) {
            tail_sum += compute_kldiv_value(log_predictions[j], targets[j]);
        }
        atomicAdd(output, tail_sum);
    }
}

torch::Tensor kl_div_cuda_forward_combined(
    torch::Tensor log_predictions,
    torch::Tensor targets) {

    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());

    const int threads = 256;
    int vec_count = n / 4;
    const int blocks = min((vec_count + threads - 1) / threads, 1024);

    kl_div_kernel_combined<<<blocks, threads>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward_combined, "Combined KL divergence forward (CUDA)");
}