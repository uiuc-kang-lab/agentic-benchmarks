#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template<typename T>
__device__ __forceinline__ T sigmoid_compute(T val) {
    T exp_val = expf(-val);
    return 1.0f / (1.0f + exp_val);
}

template<typename scalar_t>
__global__ void sigmoid_kernel_vectorized(const scalar_t* __restrict__ input,
                                         scalar_t* __restrict__ output,
                                         const int64_t size) {
    constexpr int vec_size = sizeof(float4) / sizeof(scalar_t);
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    // Process multiple vectors per thread using grid-stride loop
    for (int idx = tid; idx < size / vec_size; idx += stride) {
        int i = idx * vec_size;
        if (i < size) {
            float4 in_chunk = *reinterpret_cast<const float4*>(&input[i]);
            float4 out_chunk;
            
            out_chunk.x = sigmoid_compute(in_chunk.x);
            out_chunk.y = sigmoid_compute(in_chunk.y);
            out_chunk.z = sigmoid_compute(in_chunk.z);
            out_chunk.w = sigmoid_compute(in_chunk.w);
            
            *reinterpret_cast<float4*>(&output[i]) = out_chunk;
        }
    }
    
    // Handle remaining elements
    int remainder = size % vec_size;
    int remainder_start = size - remainder;
    if (tid < remainder) {
        output[remainder_start + tid] = sigmoid_compute(input[remainder_start + tid]);
    }
}

torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int64_t size = input.numel();

    const int threads = 256;
    int num_sms;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    const int blocks_per_sm = 2;  // Adjust based on occupancy requirements
    const int total_blocks = num_sms * blocks_per_sm;
    
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sigmoid_kernel", [&] {
        sigmoid_kernel_vectorized<scalar_t><<<total_blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size
        );
    });

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Balanced Vectorized Sigmoid forward (CUDA)");
}