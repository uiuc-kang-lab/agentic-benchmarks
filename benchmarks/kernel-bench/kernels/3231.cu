#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>

template <typename scalar_t>
__global__ void log_softmax_forward_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int dim_size) {

    const int batch_idx = blockIdx.x;
    const int tid = threadIdx.x;
    
    // Fixed block size of 128 threads optimized for H100
    constexpr int BLOCK_SIZE = 128;
    
    const scalar_t* __restrict__ input_row = input + batch_idx * dim_size;
    scalar_t* output_row = output + batch_idx * dim_size;

    // Static shared memory allocation
    __shared__ scalar_t sdata[BLOCK_SIZE];

    // Initialize max_val
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    
    // Compute max value using grid-stride loop
    #pragma unroll 4
    for (int idx = tid; idx < dim_size; idx += BLOCK_SIZE) {
        max_val = max(max_val, __ldg(input_row + idx));
    }

    sdata[tid] = max_val;
    __syncthreads();

    // Reduce max value within block
    #pragma unroll
    for (int offset = BLOCK_SIZE/2; offset > 0; offset >>= 1) {
        if (tid < offset) {
            sdata[tid] = max(sdata[tid], sdata[tid + offset]);
        }
        __syncthreads();
    }
    max_val = sdata[0];
    __syncthreads();

    // Compute sum of exp(input - max_val)
    scalar_t sum = 0;
    #pragma unroll 4
    for (int idx = tid; idx < dim_size; idx += BLOCK_SIZE) {
        scalar_t val = exp(__ldg(input_row + idx) - max_val);
        output_row[idx] = val;
        sum += val;
    }

    sdata[tid] = sum;
    __syncthreads();

    #pragma unroll
    for (int offset = BLOCK_SIZE/2; offset > 0; offset >>= 1) {
        if (tid < offset) {
            sdata[tid] += sdata[tid + offset];
        }
        __syncthreads();
    }
    sum = sdata[0];
    __syncthreads();

    scalar_t log_sum = log(sum);

    // Compute final output
    #pragma unroll 4
    for (int idx = tid; idx < dim_size; idx += BLOCK_SIZE) {
        output_row[idx] = (__ldg(input_row + idx) - max_val) - log_sum;
    }
}

torch::Tensor log_softmax_cuda_forward(torch::Tensor input, int64_t dim) {
    TORCH_CHECK(input.is_cuda(), "input must be a CUDA tensor");
    TORCH_CHECK(
        input.scalar_type() == torch::kFloat32 || input.scalar_type() == torch::kFloat64,
        "input must be float32 or float64");

    int64_t ndim = input.dim();
    TORCH_CHECK(dim >= -ndim && dim < ndim, "dim out of range");
    dim = dim >= 0 ? dim : dim + ndim;

    std::vector<int64_t> permute_dims;
    for (int64_t i = 0; i < ndim; ++i) {
        if (i != dim) permute_dims.push_back(i);
    }
    permute_dims.push_back(dim);

    input = input.permute(permute_dims).contiguous();
    int64_t batch_size = input.numel() / input.size(-1);
    int64_t dim_size = input.size(-1);

    auto output = torch::empty_like(input);

    constexpr int BLOCK_SIZE = 128;
    const int blocks = batch_size;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "log_softmax_forward_cuda", ([&] {
        log_softmax_forward_kernel<scalar_t><<<blocks, BLOCK_SIZE>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            dim_size);
    }));

    std::vector<int64_t> inverse_permute_dims(ndim);
    for (size_t i = 0; i < permute_dims.size(); ++i) {
        inverse_permute_dims[permute_dims[i]] = i;
    }
    output = output.permute(inverse_permute_dims);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &log_softmax_cuda_forward, "LogSoftmax forward (CUDA)");
}