#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized 3D Average Pooling Kernel using combined ideas from two versions:
// - Grid mapping using blockIdx.z to combine (n, c, d_out)
// - Thread block configured as (32, 8, 1) for improved memory coalescing along the width dimension
// - Pointer arithmetic precomputations for efficient inner loop over the pooling window

__global__ void avg_pool3d_forward_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int batch_size, int channels,
    int in_d, int in_h, int in_w,
    int out_d, int out_h, int out_w,
    int kernel_size, int stride, int padding) {

    // Decode the combined (n, c, d_out) from blockIdx.z
    int idx = blockIdx.z;
    int d_out = idx % out_d;
    idx /= out_d;
    int c = idx % channels;
    int n = idx / channels;

    // Compute output spatial indices using 2D grid and thread indices
    int h_out = blockIdx.y * blockDim.y + threadIdx.y;
    int w_out = blockIdx.x * blockDim.x + threadIdx.x;
    if (h_out >= out_h || w_out >= out_w) return;

    // Determine the pooling window boundaries in the input
    int d_start = d_out * stride - padding;
    int h_start = h_out * stride - padding;
    int w_start = w_out * stride - padding;
    
    // Clamp boundaries to ensure we are within valid input range
    int d_start_clamped = max(d_start, 0);
    int h_start_clamped = max(h_start, 0);
    int w_start_clamped = max(w_start, 0);
    int d_end_clamped = min(d_start + kernel_size, in_d);
    int h_end_clamped = min(h_start + kernel_size, in_h);
    int w_end_clamped = min(w_start + kernel_size, in_w);

    float sum = 0.0f;
    int pool_volume = kernel_size * kernel_size * kernel_size; // count_include_pad style division

    // Precompute base offset for the current (n, c) to save recomputation
    int baseOffset = (n * channels + c) * in_d;

    // Loop over the pooling window using unrolled loops for d and h
    #pragma unroll
    for (int d = d_start_clamped; d < d_end_clamped; d++) {
        // Compute the pointer offset for current depth slice
        int d_offset = (baseOffset + d) * in_h * in_w;
        #pragma unroll
        for (int h = h_start_clamped; h < h_end_clamped; h++) {
            // Compute the starting index for the row in the input
            int row_start = d_offset + h * in_w + w_start_clamped;
            int row_length = w_end_clamped - w_start_clamped;
            #pragma unroll
            for (int offset = 0; offset < row_length; offset++) {
                sum += input[row_start + offset];
            }
        }
    }

    // Compute the linear index for the output and store the averaged result
    int output_idx = (((n * channels + c) * out_d + d_out) * out_h + h_out) * out_w + w_out;
    output[output_idx] = sum / static_cast<float>(pool_volume);
}

at::Tensor forward(at::Tensor input, int kernel_size, int stride, int padding) {
    TORCH_CHECK(input.dim() == 5, "Input tensor must be 5-dimensional");
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    int batch_size = input.size(0);
    int channels   = input.size(1);
    int in_d       = input.size(2);
    int in_h       = input.size(3);
    int in_w       = input.size(4);

    // Calculate output dimensions based on convolution arithmetic
    int out_d = (in_d + 2 * padding - kernel_size) / stride + 1;
    int out_h = (in_h + 2 * padding - kernel_size) / stride + 1;
    int out_w = (in_w + 2 * padding - kernel_size) / stride + 1;

    auto output = at::empty({batch_size, channels, out_d, out_h, out_w}, input.options());

    // Configure thread block and grid dimensions for optimal memory access
    dim3 block(32, 8, 1);  // 32 threads in width for coalesced global memory accesses
    dim3 grid((out_w + block.x - 1) / block.x,
              (out_h + block.y - 1) / block.y,
              batch_size * channels * out_d);  // combine n, c, and d_out

    avg_pool3d_forward_kernel<<<grid, block>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size, channels,
        in_d, in_h, in_w,
        out_d, out_h, out_w,
        kernel_size, stride, padding);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized 3D Average Pooling forward (CUDA)");
}
