#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define TILE_SIZE 32
#define ALIGN_SIZE 4  // For float4 alignment

// Using double-buffering to overlap global loads with computation


__global__ void aligned_matrix_multiply_kernel(const float* __restrict__ A,
                                             const float* __restrict__ B,
                                             float* __restrict__ C,
                                             const int M, const int N, const int K) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int row = blockIdx.y * TILE_SIZE + ty;
    const int col = blockIdx.x * TILE_SIZE + tx;
    
    float sum = 0.0f;

    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < M && (t * TILE_SIZE + tx) < K) {
            int offset_A = row * K + t * TILE_SIZE + tx;
            As[ty][tx] = __ldg(&A[offset_A]);
        } else {
            As[ty][tx] = 0.0f;
        }

        if ((t * TILE_SIZE + ty) < K && col < N) {
            int offset_B = (t * TILE_SIZE + ty) * N + col;
            Bs[ty][tx] = __ldg(&B[offset_B]);
        } else {
            Bs[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[ty][k] * Bs[k][tx];
        }
        
        __syncthreads();
    }
    
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

void matrix_multiply_cuda(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const float *d_A = A.data_ptr<float>();
    const float *d_B = B.data_ptr<float>();
    float *d_C = C.data_ptr<float>();

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);

    aligned_matrix_multiply_kernel<<<grid, threads>>>(d_A, d_B, d_C, M, N, K);
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int N = B.size(1);

    torch::Tensor C = torch::empty({M, N}, A.options());
    matrix_multiply_cuda(A, B, C);
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Aligned matrix multiplication with __ldg() (CUDA)");
}