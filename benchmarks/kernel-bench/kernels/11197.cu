#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void smooth_l1_loss_kernel(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    int n_elements
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    float thread_sum = 0.0f;

    // Process 4 elements at a time
    for (int i = idx * 4; i < n_elements - 3; i += stride * 4) {
        float diff0 = predictions[i] - targets[i];
        float diff1 = predictions[i+1] - targets[i+1];
        float diff2 = predictions[i+2] - targets[i+2];
        float diff3 = predictions[i+3] - targets[i+3];

        float abs_diff0 = fabsf(diff0);
        float abs_diff1 = fabsf(diff1);
        float abs_diff2 = fabsf(diff2);
        float abs_diff3 = fabsf(diff3);

        thread_sum += (abs_diff0 < 1.0f) ? 0.5f * diff0 * diff0 : abs_diff0 - 0.5f;
        thread_sum += (abs_diff1 < 1.0f) ? 0.5f * diff1 * diff1 : abs_diff1 - 0.5f;
        thread_sum += (abs_diff2 < 1.0f) ? 0.5f * diff2 * diff2 : abs_diff2 - 0.5f;
        thread_sum += (abs_diff3 < 1.0f) ? 0.5f * diff3 * diff3 : abs_diff3 - 0.5f;
    }

    // Handle remaining elements
    for (int i = idx * 4 + (n_elements / 4 * 4); i < n_elements; i += stride) {
        float diff = predictions[i] - targets[i];
        float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;
    }

    // Block-wise reduction using warp-level primitives
    int tid = threadIdx.x;
    int lane = tid & 31;
    int warpId = tid >> 5;

    // Warp-level reduction within each warp
    for (int offset = 16; offset > 0; offset /= 2) {
        thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
    }

    // Write warp result to shared memory
    __shared__ float warp_sums[32]; // Maximum of 32 warps per block (for 1024 threads max)
    if (lane == 0) {
        warp_sums[warpId] = thread_sum;
    }
    __syncthreads();

    // Final reduction by the first warp
    if (tid < blockDim.x / 32) {
        thread_sum = warp_sums[lane];
        for (int offset = 16; offset > 0; offset /= 2) {
            thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
        }
    }
    if (tid == 0) {
        atomicAdd(output, thread_sum / n_elements);
    }
}

torch::Tensor smooth_l1_loss_cuda(
    torch::Tensor predictions,
    torch::Tensor targets
) {
    TORCH_CHECK(
        predictions.sizes() == targets.sizes(),
        "Input tensors must have the same shape"
    );
    TORCH_CHECK(
        predictions.is_contiguous() && targets.is_contiguous(),
        "Input tensors must be contiguous"
    );
    TORCH_CHECK(
        predictions.device().is_cuda() && targets.device().is_cuda(),
        "Inputs must be CUDA tensors"
    );

    int n = predictions.numel();
    auto output = torch::zeros({1}, predictions.options());

    const int block_size = 256;
    const int grid_size = (n + block_size * 4 - 1) / (block_size * 4);

    smooth_l1_loss_kernel<<<grid_size, block_size>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &smooth_l1_loss_cuda, "Smooth L1 Loss (CUDA)");
}