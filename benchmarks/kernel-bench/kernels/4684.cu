#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__constant__ int d_batch_size;
__constant__ int d_num_features;
__constant__ int d_numel_per_batch;
__constant__ float d_eps;

template <typename scalar_t>
__device__ scalar_t calculate_sumsq(
    const scalar_t* __restrict__ input,
    const int batch_offset,
    const int offset_in_batch
) {
    scalar_t sumsq = 0.0f;
    #pragma unroll 4
    for (int feat = 0; feat < d_num_features; feat++) {
        const scalar_t val = input[batch_offset + feat * d_numel_per_batch + offset_in_batch];
        sumsq += val * val;
    }
    return sumsq;
}

template <typename scalar_t>
__device__ void normalize_features(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_offset,
    const int offset_in_batch,
    const scalar_t inv_rms
) {
    #pragma unroll 4
    for (int feat = 0; feat < d_num_features; feat++) {
        const int idx = batch_offset + feat * d_numel_per_batch + offset_in_batch;
        output[idx] = input[idx] * inv_rms;
    }
}

template <typename scalar_t>
__global__ void rms_norm_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int batch_id = tid / d_numel_per_batch;
    
    if (batch_id >= d_batch_size) return;
    
    const int offset_in_batch = tid % d_numel_per_batch;
    const int batch_offset = batch_id * d_num_features * d_numel_per_batch;

    // Calculate sum of squares
    const scalar_t sumsq = calculate_sumsq(
        input, batch_offset, offset_in_batch
    );
    
    // Calculate inverse RMS for multiplication instead of division
    const scalar_t inv_rms = rsqrt(sumsq / d_num_features + d_eps);
    
    // Normalize
    normalize_features(
        input, output, batch_offset, offset_in_batch, inv_rms
    );
}

torch::Tensor rms_norm_cuda_forward(torch::Tensor input, float eps) {
    auto output = torch::empty_like(input);
    
    const int batch_size = input.size(0);
    const int num_features = input.size(1);
    
    int numel_per_batch = 1;
    for(int i = 2; i < input.dim(); i++) {
        numel_per_batch *= input.size(i);
    }

    // Copy constants to device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_batch_size), &batch_size, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_features), &num_features, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_numel_per_batch), &numel_per_batch, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_eps), &eps, sizeof(float));

    const int total_threads = batch_size * numel_per_batch;
    const int threads_per_block = 512;
    const int blocks = (total_threads + threads_per_block - 1) / threads_per_block;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "rms_norm_cuda", ([&] {
        rms_norm_kernel<scalar_t><<<blocks, threads_per_block>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>()
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &rms_norm_cuda_forward, "RMS normalization forward (CUDA)");
}