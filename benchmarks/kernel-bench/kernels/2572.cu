#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void relu_kernel_coalesced(
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ input,
    const int64_t size) {

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = gridDim.x * blockDim.x;

    // Memory coalesced processing
    #pragma unroll
    for (int i = idx; i < size; i += stride) {
        output[i] = max(input[i], static_cast<scalar_t>(0));
    }
}

// PyTorch wrapper function
torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int64_t size = input.numel();
    
    int threads = 256;  // Using 256 threads per block for optimal coalescing
    int blocks = (size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "relu_kernel_coalesced", ([&] {
        relu_kernel_coalesced<scalar_t><<<blocks, threads>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            size
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Coalesced ReLU forward (CUDA)");
}