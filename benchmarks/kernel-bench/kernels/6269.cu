#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void avg_pool3d_forward_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int batch_size, int channels,
    int in_d, int in_h, int in_w,
    int out_d, int out_h, int out_w,
    int kernel_size, int stride, int padding) {

    // Shared memory for partial sums within the block
    __shared__ float shared_data[32 * 8];
    
    // Decode the combined (n, c, d_out) from blockIdx.z
    int idx = blockIdx.z;
    int d_out = idx % out_d;
    idx /= out_d;
    int c = idx % channels;
    int n = idx / channels;

    // Thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;
    
    // Output coordinates
    int h_out = blockIdx.y * blockDim.y + ty;
    int w_out = blockIdx.x * blockDim.x + tx;
    
    if (h_out >= out_h || w_out >= out_w) return;

    // Calculate input window boundaries
    int d_start = d_out * stride - padding;
    int h_start = h_out * stride - padding;
    int w_start = w_out * stride - padding;
    
    // Clamp boundaries
    int d_start_clamped = max(d_start, 0);
    int h_start_clamped = max(h_start, 0);
    int w_start_clamped = max(w_start, 0);
    int d_end_clamped = min(d_start + kernel_size, in_d);
    int h_end_clamped = min(h_start + kernel_size, in_h);
    int w_end_clamped = min(w_start + kernel_size, in_w);

    // Initialize partial sum
    float partial_sum = 0.0f;
    
    // Base offset for current (n,c) slice
    int baseOffset = (n * channels + c) * in_d;

    // Accumulate values for this thread's output element
    #pragma unroll
    for (int d = d_start_clamped; d < d_end_clamped; d++) {
        int d_offset = (baseOffset + d) * in_h * in_w;
        #pragma unroll
        for (int h = h_start_clamped; h < h_end_clamped; h++) {
            int row_start = d_offset + h * in_w + w_start_clamped;
            #pragma unroll
            for (int w = 0; w < w_end_clamped - w_start_clamped; w++) {
                partial_sum += input[row_start + w];
            }
        }
    }

    // Compute and store final averaged result directly
    int pool_volume = kernel_size * kernel_size * kernel_size;
    int output_idx = (((n * channels + c) * out_d + d_out) * out_h + h_out) * out_w + w_out;
    output[output_idx] = partial_sum / static_cast<float>(pool_volume);

}

at::Tensor forward(at::Tensor input, int kernel_size, int stride, int padding) {
    TORCH_CHECK(input.dim() == 5, "Input tensor must be 5-dimensional");
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    int batch_size = input.size(0);
    int channels   = input.size(1);
    int in_d       = input.size(2);
    int in_h       = input.size(3);
    int in_w       = input.size(4);

    int out_d = (in_d + 2 * padding - kernel_size) / stride + 1;
    int out_h = (in_h + 2 * padding - kernel_size) / stride + 1;
    int out_w = (in_w + 2 * padding - kernel_size) / stride + 1;

    auto output = at::empty({batch_size, channels, out_d, out_h, out_w}, input.options());

    // Configure thread block and grid dimensions
    dim3 block(32, 8, 1);
    dim3 grid((out_w + block.x - 1) / block.x,
              (out_h + block.y - 1) / block.y,
              batch_size * channels * out_d);

    avg_pool3d_forward_kernel<<<grid, block>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size, channels,
        in_d, in_h, in_w,
        out_d, out_h, out_w,
        kernel_size, stride, padding);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D Average Pooling forward (CUDA) with shared memory");
}