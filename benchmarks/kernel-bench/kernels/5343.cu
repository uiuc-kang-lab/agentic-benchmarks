#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t, int KERNEL_SIZE>
__global__ void max_pool2d_tiled_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int stride,
    const int padding,
    const int dilation
) {
    const int oh = blockIdx.y * blockDim.y + threadIdx.y;
    const int ow = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (oh >= output_height || ow >= output_width) return;

    for (int b = 0; b < batch_size; b++) {
        for (int c = 0; c < channels; c++) {
            scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
            const int input_batch_offset = b * (channels * input_height * input_width);
            const int input_channel_offset = c * (input_height * input_width);

            if constexpr (KERNEL_SIZE == 2) {
                const int ih_base = oh * stride - padding;
                const int iw_base = ow * stride - padding;

                #pragma unroll
                for (int kh = 0; kh < 2; kh++) {
                    const int ih = ih_base + kh * dilation;
                    if (ih >= 0 && ih < input_height) {
                        const int ih_offset = ih * input_width;
                        #pragma unroll
                        for (int kw = 0; kw < 2; kw++) {
                            const int iw = iw_base + kw * dilation;
                            if (iw >= 0 && iw < input_width) {
                                const int idx = input_batch_offset + input_channel_offset + ih_offset + iw;
                                max_val = max(max_val, __ldg(&input[idx]));
                            }
                        }
                    }
                }
            }
            else if constexpr (KERNEL_SIZE == 3) {
                const int ih_base = oh * stride - padding;
                const int iw_base = ow * stride - padding;

                #pragma unroll
                for (int kh = 0; kh < 3; kh++) {
                    const int ih = ih_base + kh * dilation;
                    if (ih >= 0 && ih < input_height) {
                        const int ih_offset = ih * input_width;
                        #pragma unroll
                        for (int kw = 0; kw < 3; kw++) {
                            const int iw = iw_base + kw * dilation;
                            if (iw >= 0 && iw < input_width) {
                                const int idx = input_batch_offset + input_channel_offset + ih_offset + iw;
                                max_val = max(max_val, __ldg(&input[idx]));
                            }
                        }
                    }
                }
            }
            else {
                for (int kh = 0; kh < KERNEL_SIZE; kh++) {
                    const int ih = oh * stride - padding + kh * dilation;
                    if (ih >= 0 && ih < input_height) {
                        const int ih_offset = ih * input_width;
                        for (int kw = 0; kw < KERNEL_SIZE; kw++) {
                            const int iw = ow * stride - padding + kw * dilation;
                            if (iw >= 0 && iw < input_width) {
                                const int idx = input_batch_offset + input_channel_offset + ih_offset + iw;
                                max_val = max(max_val, __ldg(&input[idx]));
                            }
                        }
                    }
                }
            }

            const int output_idx = (b * channels + c) * output_height * output_width + oh * output_width + ow;
            output[output_idx] = max_val;
        }
    }
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    dim3 threads(16, 16);
    dim3 blocks(
        (output_width + threads.x - 1) / threads.x,
        (output_height + threads.y - 1) / threads.y
    );

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        if (kernel_size == 2) {
            max_pool2d_tiled_kernel<scalar_t, 2><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(),
                batch_size, channels, input_height, input_width,
                output_height, output_width, stride, padding, dilation);
        }
        else if (kernel_size == 3) {
            max_pool2d_tiled_kernel<scalar_t, 3><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(),
                batch_size, channels, input_height, input_width,
                output_height, output_width, stride, padding, dilation);
        }
        else {
            max_pool2d_tiled_kernel<scalar_t, -1><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(),
                batch_size, channels, input_height, input_width,
                output_height, output_width, stride, padding, dilation);
        }
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward with tiled execution (CUDA)");
}