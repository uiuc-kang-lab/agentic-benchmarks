#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32

__global__ void minimized_divergence_mm_kernel(const float* __restrict__ A,
                                               const float* __restrict__ B,
                                               float* __restrict__ C,
                                               int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    int bx = blockIdx.x * TILE_SIZE;
    int by = blockIdx.y * TILE_SIZE;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by + ty;
    int col = bx + tx;
    
    float sum = 0.0f;
    bool compute = (row < N && col < N);

    for (int tile = 0; tile < (N + TILE_SIZE - 1) / TILE_SIZE; ++tile) {
        if (row < N && (tile * TILE_SIZE + tx) < N) {
            As[ty][tx] = A[row * N + (tile * TILE_SIZE + tx)];
        } else {
            As[ty][tx] = 0.0f;
        }
        
        if (col < N && (tile * TILE_SIZE + ty) < N) {
            Bs[ty][tx] = B[(tile * TILE_SIZE + ty) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        if (compute) {
            for (int k = 0; k < TILE_SIZE; ++k) {
                sum += As[ty][k] * Bs[k][tx];
            }
        }
        
        __syncthreads();
    }
    
    if (compute && row >= col) {
        C[row * N + col] = sum;
    } else if (compute) {
        C[row * N + col] = 0.0f;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    minimized_divergence_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Minimized divergence triangular matrix multiplication (CUDA)");
}
