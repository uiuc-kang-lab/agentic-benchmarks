#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// This CUDA kernel performs 2D convolution on asymmetric input with a square kernel.
// It optimizes thread and block distribution to ensure even workload across threads.

__global__ void conv2d_optimized_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int out_height,
    int out_width,
    int kernel_size,
    int stride,
    int padding,
    int dilation) {

    // Calculate the global thread index
    int n = blockIdx.x;       // batch index
    int oc = blockIdx.y;      // output channel index
    int out_y = blockIdx.z * blockDim.y + threadIdx.y;
    int out_x = threadIdx.x;

    if (out_y >= out_height || out_x >= out_width) return;

    float sum = 0.0f;

    // Loop over input channels
    for (int ic = 0; ic < in_channels; ++ic) {
        // Unroll kernel height and width loops
        #pragma unroll
        for (int ky = 0; ky < kernel_size; ++ky) {
            #pragma unroll
            for (int kx = 0; kx < kernel_size; ++kx) {
                int in_y = out_y * stride - padding + ky * dilation;
                int in_x = out_x * stride - padding + kx * dilation;

                // Boundary check
                if (in_y >= 0 && in_y < in_height && in_x >= 0 && in_x < in_width) {
                    int input_idx = n * in_channels * in_height * in_width
                                  + ic * in_height * in_width
                                  + in_y * in_width
                                  + in_x;
                    int weight_idx = oc * in_channels * kernel_size * kernel_size
                                   + ic * kernel_size * kernel_size
                                   + ky * kernel_size
                                   + kx;
                    sum += input[input_idx] * weight[weight_idx];
                }
            }
        }
    }

    // Add bias if provided
    if (bias) {
        sum += bias[oc];
    }
    
    int output_idx = n * out_channels * out_height * out_width
                   + oc * out_height * out_width
                   + out_y * out_width
                   + out_x;
    output[output_idx] = sum;
}


// Host function that prepares the tensors and launches the CUDA kernel
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    // Input checks
    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    // This implementation supports groups==1 only
    TORCH_CHECK(groups == 1, "Only groups==1 is supported in this optimized kernel");

    // Extract dimensions
    const int batch = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);

    const int out_channels = weight.size(0);
    const int kernel_size = weight.size(2); // square kernel assumed (weight.size(2) == weight.size(3))
    
    // Compute output dimensions
    const int out_height = (in_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;
    const int out_width  = (in_width  + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;

    auto output = torch::zeros({batch, out_channels, out_height, out_width}, x.options());
    
    // Determine kernel launch parameters
    dim3 threads(out_width, 1);
    dim3 blocks(batch, out_channels, (out_height + threads.y - 1) / threads.y);

    const float* input_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = (bias.has_value()) ? bias.value().data_ptr<float>() : nullptr;
    float* output_ptr = output.data_ptr<float>();

    // Launch the CUDA convolution kernel
    conv2d_optimized_kernel<<<blocks, threads>>>(
        input_ptr, weight_ptr, bias_ptr, output_ptr,
        batch, in_channels, out_channels,
        in_height, in_width, out_height, out_width,
        kernel_size, stride, padding, dilation);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized CUDA forward function for 2D convolution with improved thread and block distribution");
}