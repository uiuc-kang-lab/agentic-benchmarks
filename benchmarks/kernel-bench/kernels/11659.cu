#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device function for computing KL divergence element
__device__ __forceinline__ float compute_kl_element(float log_pred, float target) {
    return expf(log_pred) - target * log_pred;
}

// Warp-level reduction using shuffle down
__device__ __forceinline__ float warp_reduce_sum(float val) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

// Main CUDA kernel using warp-level primitives
__global__ void kl_div_kernel_warp(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    // Get thread ID
    const unsigned int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;
    const int stride = blockDim.x * gridDim.x;
    
    // Compute local sum using stride loop
    float sum = 0.0f;
    for (; idx < n; idx += stride) {
        sum += compute_kl_element(log_predictions[idx], targets[idx]);
    }
    
    // Perform warp-level reduction
    sum = warp_reduce_sum(sum);
    
    // Write result from the first thread of each warp
    if (tid % warpSize == 0) {
        atomicAdd(output, sum);
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    const int threads = 256;
    const int blocks = min((n + threads - 1) / threads, 1024);
    
    kl_div_kernel_warp<<<blocks, threads>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward (CUDA)");
}
