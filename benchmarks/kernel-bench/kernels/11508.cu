#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void kl_div_kernel(const float* __restrict__ log_predictions,
                             const float* __restrict__ targets, 
                             float* __restrict__ output,
                             const int n) {
    extern __shared__ float sdata[];

    // Calculate 1D global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    float temp = 0.0f;

    // Process elements aligned for coalesced access
    while (idx < n) {
        float log_pred = log_predictions[idx];
        float target = targets[idx];
        temp += expf(log_pred) - target * log_pred;
        idx += gridDim.x * blockDim.x;
    }

    // Store thread result in shared memory
    sdata[tid] = temp;
    __syncthreads();

    // Perform reduction within the block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) atomicAdd(output, sdata[0]);
}

torch::Tensor kl_div_cuda_forward(torch::Tensor log_predictions, torch::Tensor targets) {
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;

    kl_div_kernel<<<blocks, threads, threads * sizeof(float)>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward optimized (CUDA)");
}