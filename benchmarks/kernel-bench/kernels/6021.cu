#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Store pooling parameters in constant memory for quick access
// pool_params[0] = kernel_size, pool_params[1] = stride, pool_params[2] = padding,
// pool_params[3] = input_length, pool_params[4] = output_length
__constant__ int pool_params[5];

// Define macros for ease of use
#define KERNEL_SIZE   (pool_params[0])
#define STRIDE        (pool_params[1])
#define PADDING       (pool_params[2])
#define INPUT_LENGTH  (pool_params[3])
#define OUTPUT_LENGTH (pool_params[4])

// Efficient kernel using a grid-stride loop for flexible parallelization
__global__ void avg_pool1d_kernel(
    const float *input,
    float *output,
    int batch_size,
    int in_channels) {

    // Shared memory for input caching
    extern __shared__ float shared_input[];
    
    // Total number of output elements
    int totalElements = batch_size * in_channels * OUTPUT_LENGTH;
    
    // Process multiple elements per thread for better arithmetic intensity
    constexpr int ELEMENTS_PER_THREAD = 4;
    
    for (int base_index = blockIdx.x * blockDim.x * ELEMENTS_PER_THREAD + threadIdx.x;
         base_index < totalElements;
         base_index += blockDim.x * gridDim.x * ELEMENTS_PER_THREAD) {
             
        for (int e = 0; e < ELEMENTS_PER_THREAD; e++) {
            int index = base_index + e * blockDim.x;
            if (index >= totalElements) break;

            // Decompose linear index into batch, channel, and output index
            int tmp = index;
            int o = tmp % OUTPUT_LENGTH;
            tmp /= OUTPUT_LENGTH;
            int channel = tmp % in_channels;
            int batch = tmp / in_channels;

            // Compute starting index for pooling window considering padding
            int start = o * STRIDE - PADDING;
            float sum = 0.0f;
            
            // Load input segment into shared memory
            int shared_offset = threadIdx.x * KERNEL_SIZE;
            for (int k = 0; k < KERNEL_SIZE; k += blockDim.x) {
                int pos = start + k;
                if (k + threadIdx.x < KERNEL_SIZE && pos >= 0 && pos < INPUT_LENGTH) {
                    int input_idx = batch * in_channels * INPUT_LENGTH + channel * INPUT_LENGTH + pos;
                    shared_input[shared_offset + k] = input[input_idx];
                } else {
                    shared_input[shared_offset + k] = 0.0f;
                }
            }
            __syncthreads();
            
            // Sum over the pooling window using shared memory
            for (int k = 0; k < KERNEL_SIZE; k++) {
                int pos = start + k;
                if (pos >= 0 && pos < INPUT_LENGTH) {
                    sum += shared_input[shared_offset + k];
                }
            }
            __syncthreads();

            // Write the averaged result
            if (index < totalElements) {
                output[index] = sum / KERNEL_SIZE;
            }
        }
    }
}

// Host function for average pooling forward pass
torch::Tensor avg_pool1d_forward(
    const torch::Tensor &x,
    int kernel_size,
    int stride,
    int padding) {

    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(kernel_size > 0 && stride > 0 && padding >= 0, "Invalid kernel parameters");

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int input_length = x.size(2);
    int output_length = (input_length + 2 * padding - kernel_size) / stride + 1;

    // Copy parameters to constant memory for fast access
    int h_pool_params[5] = { kernel_size, stride, padding, input_length, output_length };
    hipMemcpyToSymbol(HIP_SYMBOL(pool_params), h_pool_params, 5 * sizeof(int));

    auto output = torch::empty({batch_size, in_channels, output_length}, x.options());

    int totalElements = batch_size * in_channels * output_length;
    int threads = 256;
    int blocks = (totalElements + threads - 1) / threads;

    avg_pool1d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size,
        in_channels
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &avg_pool1d_forward, "1D Average Pooling forward with constant memory and grid-stride loop (CUDA)");
}
