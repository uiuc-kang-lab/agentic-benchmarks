#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <iostream>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void matmul_warp_level(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;
    for (int i = 0; i < K; ++i) {
        float a = A[row * K + i];
        float b = B[i * N + col];
        sum += a * b;
    }

    // Warp-level reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
    }

    // Write the result for each warp's first thread
    if (threadIdx.x % warpSize == 0) {
        C[row * N + col] = sum;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((N + 31) / 32, (M + 31) / 32);

    matmul_warp_level<<<blocksPerGrid, threadsPerBlock>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, N, K);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication with warp-level primitives (CUDA)");
}