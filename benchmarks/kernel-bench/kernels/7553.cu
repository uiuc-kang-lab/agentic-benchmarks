#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Combined kernel: uses warp-level parallel reduction (from Kernel 1) to parallelize the inner summation
// and uses a simple output index decomposition (from Kernel 2) to improve memory coalescing of output assignments.
// This kernel assigns one warp per output element. Each lane in the warp processes a subset of the accumulated contributions
// (over input channels and kernel volume) and then the warp performs an efficient reduction using __shfl_down_sync.


template <typename scalar_t>
__global__ void transposed_conv3d_combined_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,  // can be nullptr
    scalar_t* __restrict__ output,
    // Input dimensions
    int N, int in_channels, int in_depth, int in_height, int in_width,
    // Output dimensions
    int out_channels, int out_depth, int out_height, int out_width,
    // Kernel dimensions
    int kT, int kH, int kW,
    // Stride
    int stride_d, int stride_h, int stride_w,
    // Padding
    int pad_d, int pad_h, int pad_w,
    // Output padding
    int out_pad_d, int out_pad_h, int out_pad_w,
    // Groups
    int groups
) {
    const int warpSize = 32;
    int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = global_thread_id / warpSize;  // one warp computes one output element
    int lane = global_thread_id % warpSize;

    // Total number of output elements
    int total_output = N * out_channels * out_depth * out_height * out_width;
    if (warp_id >= total_output) return;

    // Decode warp_id into (n, c, d, h, w) using a memory-coalesced indexing pattern (see Kernel 2).
    int w_out = warp_id % out_width;
    int tmp = warp_id / out_width;
    int h_out = tmp % out_height;
    tmp /= out_height;
    int d_out = tmp % out_depth;
    tmp /= out_depth;
    int c_out = tmp % out_channels;
    int n = tmp / out_channels;

    // Determine group assignment and local channel index
    int group_size = out_channels / groups;
    int group = c_out / group_size;
    int out_c_local = c_out % group_size;

    // Number of input channels per group
    int in_channels_per_group = in_channels / groups;
    // Total number of iterations: for each input channel in the group and for each kernel element
    int total_iters = in_channels_per_group * (kT * kH * kW);

    scalar_t sum = 0;
    // Each lane iterates over portions of the flattened loop
    for (int i = lane; i < total_iters; i += warpSize) {
        // Decode the flattened iteration index into (ic, kd, kh, kw)
        int ic = i / (kT * kH * kW);
        int rem = i % (kT * kH * kW);
        int kd = rem / (kH * kW);
        int rem2 = rem % (kH * kW);
        int kh = rem2 / kW;
        int kw = rem2 % kW;

        // Map to actual input channel
        int input_channel = group * in_channels_per_group + ic;

        // Compute the corresponding input spatial indices based on transposed convolution arithmetic
        int d_in_tmp = d_out + pad_d - kd;
        if (d_in_tmp % stride_d != 0) continue;
        int d_in = d_in_tmp / stride_d;
        if (d_in < 0 || d_in >= in_depth) continue;

        int h_in_tmp = h_out + pad_h - kh;
        if (h_in_tmp % stride_h != 0) continue;
        int h_in = h_in_tmp / stride_h;
        if (h_in < 0 || h_in >= in_height) continue;

        int w_in_tmp = w_out + pad_w - kw;
        if (w_in_tmp % stride_w != 0) continue;
        int w_in = w_in_tmp / stride_w;
        if (w_in < 0 || w_in >= in_width) continue;

        // Compute flat index for input tensor: [N, in_channels, in_depth, in_height, in_width]
        int input_idx = (((n * in_channels + input_channel) * in_depth + d_in) * in_height + h_in) * in_width + w_in;

        // Compute flat index for weight tensor: [in_channels, out_channels/groups, kT, kH, kW]
        int weight_idx = ((((input_channel) * group_size + out_c_local) * kT + kd) * kH + kh) * kW + kw;

        // Use __ldg for read-only cache optimization
        sum += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
    }

    // Perform warp-level reduction using shuffle instructions
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    // The first lane of the warp writes the output
    if (lane == 0) {
        if (bias != nullptr) {
            sum += __ldg(&bias[c_out]);
        }
        output[warp_id] = sum;
    }
}


// Host launcher function: combines aspects from both kernel implementations
// It computes output dimensions using the transposed convolution formula and launches one warp per output element.
// Total threads launched = total_output * warpSize, with a typical block size (e.g., 256 threads).

torch::Tensor forward(
    torch::Tensor input,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> output_padding,
    int64_t groups
) {
    // Ensure that input tensors are contiguous
    input = input.contiguous();
    weight = weight.contiguous();
    torch::Tensor bias_tensor;
    if (bias.has_value()) {
        bias_tensor = bias.value().contiguous();
    }

    // Input dimensions
    int N = input.size(0);
    int in_channels = input.size(1);
    int in_depth = input.size(2);
    int in_height = input.size(3);
    int in_width = input.size(4);

    // Kernel dimensions
    int kT = weight.size(2);
    int kH = weight.size(3);
    int kW = weight.size(4);

    // out_channels: weight shape is [in_channels, out_channels/groups, kT, kH, kW]
    int out_channels = weight.size(1) * groups;

    // Compute output spatial dimensions using the transposed convolution formula
    int out_depth = (in_depth - 1) * stride[0] - 2 * padding[0] + kT + output_padding[0];
    int out_height = (in_height - 1) * stride[1] - 2 * padding[1] + kH + output_padding[1];
    int out_width  = (in_width - 1) * stride[2] - 2 * padding[2] + kW + output_padding[2];

    auto output = torch::zeros({N, out_channels, out_depth, out_height, out_width}, input.options());

    // One warp (32 threads) computes one output element
    int total_output = N * out_channels * out_depth * out_height * out_width;
    const int warpSize = 32;
    int total_threads = total_output * warpSize;
    int threads = 256;
    int blocks = (total_threads + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "transposed_conv3d_combined_kernel", ([&] {
        transposed_conv3d_combined_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            bias.has_value() ? bias_tensor.data_ptr<scalar_t>() : nullptr,
            output.data_ptr<scalar_t>(),
            N, in_channels, in_depth, in_height, in_width,
            out_channels, out_depth, out_height, out_width,
            kT, kH, kW,
            stride[0], stride[1], stride[2],
            padding[0], padding[1], padding[2],
            output_padding[0], output_padding[1], output_padding[2],
            groups
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Combined ConvTranspose3d forward with warp-level reduction and coalesced indexing",
          py::arg("input"),
          py::arg("weight"),
          py::arg("bias") = nullptr,
          py::arg("stride"),
          py::arg("padding"),
          py::arg("output_padding"),
          py::arg("groups"));
}
