#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Templated kernel specialized for common 3x3x3 kernel size with manual unrolling
template <int KD, int KH, int KW>
__global__ void conv_transposed_3d_manual_unroll_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int totalElements,
    int N, int C_in, int D_in, int H_in, int W_in,
    int C_out,
    int stride_d, int stride_h, int stride_w,
    int pad_d, int pad_h, int pad_w,
    int D_out, int H_out, int W_out,
    int groups
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    while (idx < totalElements) {
        // Decode flat index into (n, c_out, d, h, w)
        int w = idx % W_out;
        int tmp = idx / W_out;
        int h = tmp % H_out;
        tmp /= H_out;
        int d = tmp % D_out;
        tmp /= D_out;
        int c_out = tmp % C_out;
        tmp /= C_out;
        int n = tmp;

        int out_ch_per_group = C_out / groups;
        int group = c_out / out_ch_per_group;
        int c_out_in_group = c_out - group * out_ch_per_group;
        int in_ch_per_group = C_in / groups;

        float out_val = (bias != nullptr) ? bias[c_out] : 0.0f;

        // Manually unrolled loops over kernel dimensions
        #pragma unroll
        for (int r = 0; r < KD; r++) {
            int d_in_calc = d + pad_d - r;
            if (d_in_calc % stride_d != 0) continue;
            int d_in = d_in_calc / stride_d;
            if (d_in < 0 || d_in >= D_in) continue;
            
            #pragma unroll
            for (int s = 0; s < KH; s++) {
                int h_in_calc = h + pad_h - s;
                if (h_in_calc % stride_h != 0) continue;
                int h_in = h_in_calc / stride_h;
                if (h_in < 0 || h_in >= H_in) continue;
                
                #pragma unroll
                for (int t = 0; t < KW; t++) {
                    int w_in_calc = w + pad_w - t;
                    if (w_in_calc % stride_w != 0) continue;
                    int w_in = w_in_calc / stride_w;
                    if (w_in < 0 || w_in >= W_in) continue;
                    
                    // Unroll accumulation over input channels in this group
                    #pragma unroll
                    for (int c = 0; c < in_ch_per_group; c++) {
                        int actual_c = group * in_ch_per_group + c;
                        int input_idx = (((n * C_in + actual_c) * D_in + d_in) * H_in + h_in) * W_in + w_in;
                        int weight_idx = ((actual_c * out_ch_per_group + c_out_in_group) * (KD * KH * KW))
                                          + (r * KH * KW + s * KW + t);
                        out_val += input[input_idx] * weight[weight_idx];
                    }
                }
            }
        }

        output[idx] = out_val;
        idx += blockDim.x * gridDim.x;
    }
}

// Generic kernel for other kernel sizes (without compile-time unrolling)
__global__ void conv_transposed_3d_generic_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int totalElements,
    int N, int C_in, int D_in, int H_in, int W_in,
    int C_out,
    int kD, int kH, int kW,
    int stride_d, int stride_h, int stride_w,
    int pad_d, int pad_h, int pad_w,
    int D_out, int H_out, int W_out,
    int groups
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    while (idx < totalElements) {
        int w = idx % W_out;
        int tmp = idx / W_out;
        int h = tmp % H_out;
        tmp /= H_out;
        int d = tmp % D_out;
        tmp /= D_out;
        int c_out = tmp % C_out;
        tmp /= C_out;
        int n = tmp;

        int out_ch_per_group = C_out / groups;
        int group = c_out / out_ch_per_group;
        int c_out_in_group = c_out - group * out_ch_per_group;
        int in_ch_per_group = C_in / groups;

        float out_val = (bias != nullptr) ? bias[c_out] : 0.0f;

        for (int r = 0; r < kD; r++) {
            int d_in_calc = d + pad_d - r;
            if (d_in_calc % stride_d != 0) continue;
            int d_in = d_in_calc / stride_d;
            if (d_in < 0 || d_in >= D_in) continue;
            
            for (int s = 0; s < kH; s++) {
                int h_in_calc = h + pad_h - s;
                if (h_in_calc % stride_h != 0) continue;
                int h_in = h_in_calc / stride_h;
                if (h_in < 0 || h_in >= H_in) continue;
                
                for (int t = 0; t < kW; t++) {
                    int w_in_calc = w + pad_w - t;
                    if (w_in_calc % stride_w != 0) continue;
                    int w_in = w_in_calc / stride_w;
                    if (w_in < 0 || w_in >= W_in) continue;
                    
                    for (int c = 0; c < in_ch_per_group; c++) {
                        int actual_c = group * in_ch_per_group + c;
                        int input_idx = (((n * C_in + actual_c) * D_in + d_in) * H_in + h_in) * W_in + w_in;
                        int weight_idx = ((actual_c * out_ch_per_group + c_out_in_group) * (kD * kH * kW))
                                          + (r * kH * kW + s * kW + t);
                        out_val += input[input_idx] * weight[weight_idx];
                    }
                }
            }
        }

        output[idx] = out_val;
        idx += blockDim.x * gridDim.x;
    }
}

// Host forward function
torch::Tensor forward(
    torch::Tensor input,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> output_padding,
    int64_t groups
) {
    // Input dimensions
    const int N = input.size(0);
    const int C_in = input.size(1);
    const int D_in = input.size(2);
    const int H_in = input.size(3);
    const int W_in = input.size(4);

    // Kernel dimensions
    const int kD = weight.size(2);
    const int kH = weight.size(3);
    const int kW = weight.size(4);

    // Stride, padding, and output padding
    const int stride_d = stride[0];
    const int stride_h = stride[1];
    const int stride_w = stride[2];
    
    const int pad_d = padding[0];
    const int pad_h = padding[1];
    const int pad_w = padding[2];
    
    const int out_pad_d = output_padding[0];
    const int out_pad_h = output_padding[1];
    const int out_pad_w = output_padding[2];

    // Compute output dimensions (assuming dilation = 1)
    const int D_out = (D_in - 1) * stride_d - 2 * pad_d + kD + out_pad_d;
    const int H_out = (H_in - 1) * stride_h - 2 * pad_h + kH + out_pad_h;
    const int W_out = (W_in - 1) * stride_w - 2 * pad_w + kW + out_pad_w;

    const int output_channels_per_group = weight.size(1);
    const int C_out = output_channels_per_group * groups;

    auto output = torch::zeros({N, C_out, D_out, H_out, W_out}, input.options());
    int totalElements = N * C_out * D_out * H_out * W_out;
    int blockSize = 256;
    int gridSize = (totalElements + blockSize - 1) / blockSize;

    if (kD == 3 && kH == 3 && kW == 3) {
        // Launch specialized kernel with manual unrolling
        conv_transposed_3d_manual_unroll_kernel<3, 3, 3><<<gridSize, blockSize>>>(
            input.data_ptr<float>(),
            weight.data_ptr<float>(),
            bias.has_value() ? bias.value().data_ptr<float>() : nullptr,
            output.data_ptr<float>(),
            totalElements,
            N, C_in, D_in, H_in, W_in,
            C_out,
            stride_d, stride_h, stride_w,
            pad_d, pad_h, pad_w,
            D_out, H_out, W_out,
            groups
        );
    } else {
        // Fallback to generic kernel
        conv_transposed_3d_generic_kernel<<<gridSize, blockSize>>>(
            input.data_ptr<float>(),
            weight.data_ptr<float>(),
            bias.has_value() ? bias.value().data_ptr<float>() : nullptr,
            output.data_ptr<float>(),
            totalElements,
            N, C_in, D_in, H_in, W_in,
            C_out,
            kD, kH, kW,
            stride_d, stride_h, stride_w,
            pad_d, pad_h, pad_w,
            D_out, H_out, W_out,
            groups
        );
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ConvTranspose3d forward function with manual loop unrolling");
}
