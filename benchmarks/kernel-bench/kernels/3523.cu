#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Device helper: define an inline exponential function for float and double.
template <typename scalar_t>
__device__ inline scalar_t my_exp(scalar_t x);

template <>
__device__ inline float my_exp<float>(float x) {
    return expf(x);
}

template <>
__device__ inline double my_exp<double>(double x) {
    return exp(x);
}

// CUDA kernel that evenly distributes the workload among threads.
// Each thread computes a contiguous segment of the data based on its global thread ID.
// This ensures balanced workload and minimizes underutilization or bottlenecks.

template <typename scalar_t>
__global__ void selu_kernel_even_load_balance(const scalar_t* __restrict__ input,
                                               scalar_t* __restrict__ output,
                                               size_t numel) {
    // Compute a unique thread ID
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_threads = gridDim.x * blockDim.x;
    
    // Calculate the number of elements per thread and the residue
    size_t base = numel / total_threads;
    size_t residue = numel % total_threads;
    
    // Each thread processes base elements, plus one extra if its ID is less than the residue
    size_t start = tid * base + (tid < residue ? tid : residue);
    size_t count = base + (tid < residue ? 1 : 0);
    size_t end = start + count;

    for (size_t i = start; i < end; i++) {
        // Load input using __ldg for potential caching benefits
        scalar_t x = __ldg(&input[i]);
        scalar_t res = (x > static_cast<scalar_t>(0))
                           ? x
                           : static_cast<scalar_t>(1.67326324235437728481) *
                                 (my_exp(x) - static_cast<scalar_t>(1));
        output[i] = static_cast<scalar_t>(1.05070098735548049342) * res;
    }
}

// Host function launching the kernel

torch::Tensor selu_forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    auto output = torch::empty_like(input);
    size_t numel = input.numel();

    // Launch configuration: using 1024 threads per block.
    const int threads = 1024;
    int blocks = (numel + threads - 1) / threads;  // Ensures enough threads are launched

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "selu_forward_even_load_balance_cuda", ([&] {
        const scalar_t* input_ptr = input.data_ptr<scalar_t>();
        scalar_t* output_ptr = output.data_ptr<scalar_t>();
        selu_kernel_even_load_balance<scalar_t><<<blocks, threads>>>(input_ptr, output_ptr, numel);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &selu_forward, "SELU Activation Forward with Even Load Balancing (CUDA)");
}
