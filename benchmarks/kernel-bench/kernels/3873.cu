#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void softsign_kernel_optimized(const float* x, float* out, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        out[idx] = x[idx] / (1.0f + fabsf(x[idx]));
    }
}

torch::Tensor forward(torch::Tensor x) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int num_elements = x.numel();
    int threads = 256; // Optimized number of threads per block
    int blocks = (num_elements + threads - 1) / threads; // Calculate number of blocks

    softsign_kernel_optimized<<<blocks, threads>>>(
        x.data_ptr<float>(), out.data_ptr<float>(), num_elements
    );

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Softsign activation with optimized indexing (CUDA)");
}