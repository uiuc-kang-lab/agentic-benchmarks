#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <vector>

namespace py = pybind11;

__global__ void conv_transpose2d_kernel(
    const float* input,
    const float* weight,
    float* output,
    const int batch_size,
    const int in_channels,
    const int out_channels,
    const int in_height,
    const int in_width,
    const int kernel_height,
    const int kernel_width,
    const int out_height,
    const int out_width,
    const int stride_h,
    const int stride_w,
    const int pad_h,
    const int pad_w
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = batch_size * out_channels * out_height * out_width;
    
    if (idx < total_elements) {
        int w_out = idx % out_width;
        int h_out = (idx / out_width) % out_height;
        int c_out = (idx / (out_width * out_height)) % out_channels;
        int b = idx / (out_width * out_height * out_channels);
        
        float sum = 0.0f;
        
        #pragma unroll 4
        for (int c_in = 0; c_in < in_channels; ++c_in) {
            #pragma unroll
            for (int kh = 0; kh < kernel_height; ++kh) {
                #pragma unroll
                for (int kw = 0; kw < kernel_width; ++kw) {
                    int h_in = (h_out + pad_h - kh) / stride_h;
                    int w_in = (w_out + pad_w - kw) / stride_w;
                    
                    if (h_in >= 0 && h_in < in_height && w_in >= 0 && w_in < in_width) {
                        int input_idx = ((b * in_channels + c_in) * in_height + h_in) * in_width + w_in;
                        int weight_idx = ((c_out * in_channels + c_in) * kernel_height + kh) * kernel_width + kw;
                        
                        sum += input[input_idx] * weight[weight_idx];
                    }
                }
            }
        }
        output[idx] = sum;
    }
}

torch::Tensor conv_transpose2d_forward(
    torch::Tensor x,
    torch::Tensor weight,
    py::object bias_obj,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding
) {
    auto input = x.contiguous();
    auto weights = weight.contiguous();
    
    auto batch_size = input.size(0);
    auto in_channels = input.size(1);
    auto in_height = input.size(2);
    auto in_width = input.size(3);
    auto out_channels = weight.size(1);
    auto kernel_height = weight.size(2);
    auto kernel_width = weight.size(3);
    
    auto out_height = (in_height - 1) * stride[0] - 2 * padding[0] + kernel_height;
    auto out_width = (in_width - 1) * stride[1] - 2 * padding[1] + kernel_width;
    
    auto output = torch::zeros({batch_size, out_channels, out_height, out_width},
                              input.options());
    
    const int threads = 256;
    const int blocks = (batch_size * out_channels * out_height * out_width + threads - 1) / threads;
    
    conv_transpose2d_kernel<<<blocks, threads>>>(
        input.data_ptr<float>(),
        weights.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        out_channels,
        in_height,
        in_width,
        kernel_height,
        kernel_width,
        out_height,
        out_width,
        stride[0],
        stride[1],
        padding[0],
        padding[1]
    );
    
    if (!bias_obj.is_none()) {
        auto bias = bias_obj.cast<torch::Tensor>();
        output.add_(bias.view({1, out_channels, 1, 1}));
    }
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward, "Conv Transpose 2D forward",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride"),
          py::arg("padding"));
}