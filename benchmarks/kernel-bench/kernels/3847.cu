#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

// Device function to compute softplus in a numerically stable way
template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_softplus(scalar_t x) {
    if (x > 20.0) {
        return x;
    } else if (x < -20.0) {
        return exp(x);
    } else {
        return log1p(exp(x));
    }
}

// Kernel using shared memory for reduction
template <typename scalar_t>
__global__ void softplus_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < size; i += stride) {
        output[i] = compute_softplus(input[i]);
    }
}

// CUDA forward function
torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    const int threads = 256;
    const int blocks = (size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "softplus_forward_cuda", ([&] {
        softplus_kernel<scalar_t><<<blocks, threads, threads * sizeof(scalar_t)>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}