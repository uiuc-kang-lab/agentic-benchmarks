#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel leveraging warp-level primitives with manual loop unrolling using #pragma unroll
// Each warp computes one output element by summing over chunks of the reduction dimension, unrolling 4 iterations at a time.

template <typename scalar_t>
__global__ void unroll_warp_reduce_sum_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t reduce_size,
    int64_t inner_size,
    int64_t total_outputs) {

    const int warpSize = 32;
    int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = global_thread_id / warpSize;
    int lane = global_thread_id % warpSize;

    int total_warps = (gridDim.x * blockDim.x) / warpSize;

    // Each warp processes one output element in a grid-stride loop
    for (int out_idx = warp_id; out_idx < total_outputs; out_idx += total_warps) {
        // Map 1D output index to outer and inner indices
        int outer_idx = out_idx / inner_size;
        int inner_idx = out_idx % inner_size;

        // Calculate base index into the reduction dimension
        int64_t base = outer_idx * reduce_size * inner_size + inner_idx;
        scalar_t sum_val = 0;

        int i;
        // Manually unroll the loop with a factor of 4
        #pragma unroll
        for (i = lane; i <= reduce_size - warpSize * 4; i += warpSize * 4) {
            sum_val += input[base + i * inner_size] 
                     + input[base + (i + warpSize) * inner_size] 
                     + input[base + (i + 2 * warpSize) * inner_size] 
                     + input[base + (i + 3 * warpSize) * inner_size];
        }
        #pragma unroll
        for (; i < reduce_size; i += warpSize) {
            sum_val += input[base + i * inner_size];
        }

        // Warp-level reduction using shuffle down
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            sum_val += __shfl_down_sync(0xFFFFFFFF, sum_val, offset);
        }

        // Lane 0 writes the result
        if (lane == 0) {
            output[out_idx] = sum_val;
        }
    }
}

// CUDA wrapper function
torch::Tensor sum_reduce_cuda(torch::Tensor input, int64_t dim) {
    // Adjust for negative dimensions
    if (dim < 0) dim += input.dim();

    auto sizes = input.sizes().vec();
    int64_t reduce_size = sizes[dim];

    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }

    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }

    // The output tensor has the reduction dimension set to 1
    sizes[dim] = 1;
    auto output = torch::empty(sizes, input.options());

    // Total number of output elements
    int64_t total_outputs = outer_size * inner_size;

    // Each output is computed by a warp (32 threads)
    const int warpSize = 32;
    int total_threads = total_outputs * warpSize;

    // Launch configuration: choose 256 threads per block
    int threads = 256;
    int blocks = (total_threads + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sum_reduce_cuda", ([&] {
        unroll_warp_reduce_sum_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            reduce_size,
            inner_size,
            total_outputs
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &sum_reduce_cuda, "Sum reduction forward (CUDA) with manual loop unrolling");
}
