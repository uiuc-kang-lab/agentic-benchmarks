#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   int N) {
    const int warp_size = 32;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int lane_id = threadIdx.x % warp_size;
    
    if (row < N && col < N) {
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            float sum = 0.f;
            // Process elements in chunks of warp_size
            for (int k = col + lane_id; k <= row; k += warp_size) {
                if (k <= row) {
                    sum += A[row * N + k] * B[k * N + col];
                }
            }
            
            // Warp reduction using shuffle
            #pragma unroll
            for (int offset = warp_size/2; offset > 0; offset /= 2) {
                sum += __shfl_down_sync(0xffffffff, sum, offset);
            }
            
            // Only the first thread in the warp writes the result
            if (lane_id == 0) {
                C[row * N + col] = sum;
            }
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Adjust block size to ensure proper warp alignment
    const int threads_x = 32;  // One warp
    const int threads_y = 8;   // Multiple warps per block
    dim3 threadsPerBlock(threads_x, threads_y);
    dim3 numBlocks((N + threads_x - 1) / threads_x, 
                  (N + threads_y - 1) / threads_y);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}