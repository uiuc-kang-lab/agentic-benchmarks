#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define NUM_STREAMS 4

template<unsigned int blockSize>
__device__ __forceinline__ void warpReduce(volatile float* sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

__global__ void kl_div_kernel_stream(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets,
    float* __restrict__ stream_results,
    const int chunk_size,
    const int offset) {
    
    extern __shared__ float sdata[];
    const unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 8 + tid + offset;
    const unsigned int stride = blockDim.x * gridDim.x;
    const unsigned int end = offset + chunk_size;
    
    float thread_sum = 0.0f;
    
    // Process aligned elements using float4
    float4* log_pred_vec = (float4*)log_predictions;
    float4* target_vec = (float4*)targets;
    
    while (i + 7 * blockDim.x < end) {
        #pragma unroll
        for (int j = 0; j < 2; j++) {
            int vec_idx = (i + j * 4 * blockDim.x) / 4;
            float4 log_pred4 = log_pred_vec[vec_idx];
            float4 target4 = target_vec[vec_idx];
            
            thread_sum += __expf(log_pred4.x) - target4.x * log_pred4.x;
            thread_sum += __expf(log_pred4.y) - target4.y * log_pred4.y;
            thread_sum += __expf(log_pred4.z) - target4.z * log_pred4.z;
            thread_sum += __expf(log_pred4.w) - target4.w * log_pred4.w;
        }
        i += stride * 8;
    }
    
    // Handle remaining elements
    while (i < end) {
        float log_pred = log_predictions[i];
        float target = targets[i];
        thread_sum += __expf(log_pred) - target * log_pred;
        i += stride;
    }
    
    sdata[tid] = thread_sum;
    __syncthreads();
    
    if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockDim.x >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    
    if (tid < 32) warpReduce<256>(sdata, tid);
    
    if (tid == 0) {
        atomicAdd(stream_results + blockIdx.x, sdata[0]);
    }
}

__global__ void final_reduction_kernel(
    float* __restrict__ stream_results,
    float* __restrict__ output,
    const int num_blocks,
    const float normalizer) {
    
    extern __shared__ float sdata[];
    const unsigned int tid = threadIdx.x;
    
    float sum = 0.0f;
    for (int i = tid; i < num_blocks * NUM_STREAMS; i += blockDim.x) {
        sum += stream_results[i];
    }
    
    sdata[tid] = sum;
    __syncthreads();
    
    if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockDim.x >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    
    if (tid < 32) warpReduce<256>(sdata, tid);
    
    if (tid == 0) {
        output[0] = sdata[0] * normalizer;
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    const int chunk_size = (n + NUM_STREAMS - 1) / NUM_STREAMS;
    
    auto output = torch::zeros({1}, log_predictions.options());
    auto stream_results = torch::zeros({NUM_STREAMS * 256}, log_predictions.options());
    
    const int threads = 256;
    const int blocks_per_stream = min((chunk_size + threads * 8 - 1) / (threads * 8), 256);
    const float normalizer = 1.0f / static_cast<float>(n);
    
    // Create CUDA streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }
    
    // Launch kernels in different streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        const int offset = i * chunk_size;
        kl_div_kernel_stream<<<blocks_per_stream, threads, threads * sizeof(float), streams[i]>>>(
            log_predictions.data_ptr<float>(),
            targets.data_ptr<float>(),
            stream_results.data_ptr<float>(),
            chunk_size,
            offset
        );
    }
    
    // Final reduction across all streams
    final_reduction_kernel<<<1, threads, threads * sizeof(float)>>>(
        stream_results.data_ptr<float>(),
        output.data_ptr<float>(),
        blocks_per_stream,
        normalizer
    );
    
    // Cleanup streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward (CUDA)");
}