#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 16

// Utility macros to check tensor properties
#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// This kernel leverages shared memory for weight tiles and combines coalesced access and balanced workload for all threads.
__global__ void conv2d_efficient_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,  // may be nullptr if not provided
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int in_height,
    int in_width,
    int out_channels,
    int kernel_height,
    int kernel_width,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation,
    int groups) {

    __shared__ float shared_weights[BLOCK_HEIGHT][BLOCK_WIDTH];

    // Map blockIdx.z to batch and output channel indices
    int linear_idx = blockIdx.z;
    int b = linear_idx / out_channels;
    int oc = linear_idx % out_channels;

    int ow = blockIdx.x * blockDim.x + threadIdx.x;
    int oh = blockIdx.y * blockDim.y + threadIdx.y;

    if (ow >= out_width || oh >= out_height) return;

    float sum = 0.0f;

    // Determine group information
    int group_out_channels = out_channels / groups;
    int group = oc / group_out_channels;
    int in_channels_per_group = in_channels / groups;

    for (int c = 0; c < in_channels_per_group; ++c) {
        int ic = group * in_channels_per_group + c;

        for (int kh = 0; kh < kernel_height; ++kh) {
            for (int kw = 0; kw < kernel_width; ++kw) {
                if (threadIdx.x < kernel_width && threadIdx.y < kernel_height) {
                    shared_weights[threadIdx.y][threadIdx.x] = weight[(((oc * in_channels_per_group + c) * kernel_height) + threadIdx.y) * kernel_width + threadIdx.x];
                }
                __syncthreads();

                int in_y = oh * stride - padding + kh * dilation;
                int in_x = ow * stride - padding + kw * dilation;
                if (in_y >= 0 && in_y < in_height && in_x >= 0 && in_x < in_width) {
                    int input_idx = ((b * in_channels + ic) * in_height + in_y) * in_width + in_x;
                    sum += input[input_idx] * shared_weights[threadIdx.y][threadIdx.x];
                }
            }
        }
    }

    if (bias != nullptr) {
        sum += bias[oc];
    }

    int output_idx = ((b * out_channels + oc) * out_height + oh) * out_width + ow;
    output[output_idx] = sum;
}

// Forward function that configures grid and block dimensions maximally utilizing both coalesced memory access and workload balance.
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int in_height = x.size(2);
    int in_width = x.size(3);
    int out_channels = weight.size(0);
    int kernel_height = weight.size(2);
    int kernel_width = weight.size(3);

    int out_height = (in_height + 2 * padding - dilation * (kernel_height - 1) - 1) / stride + 1;
    int out_width = (in_width + 2 * padding - dilation * (kernel_width - 1) - 1) / stride + 1;

    auto options = x.options();
    auto output = torch::empty({batch_size, out_channels, out_height, out_width}, options);

    dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
    dim3 grid(
        (out_width + block.x - 1) / block.x,
        (out_height + block.y - 1) / block.y,
        batch_size * out_channels);

    const float* input_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();
    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        bias_ptr = bias.value().data_ptr<float>();
    }

    conv2d_efficient_kernel<<<grid, block>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch_size,
        in_channels,
        in_height,
        in_width,
        out_channels,
        kernel_height,
        kernel_width,
        out_height,
        out_width,
        stride,
        padding,
        dilation,
        groups
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Efficient CUDA 2D Convolution Kernel");
}
