#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <vector>

namespace py = pybind11;

// Use constant memory for weights (max 64KB)
__constant__ float c_weight[16384];

// This kernel computes one output element per block and uses loop unrolling on the inner loops
// to reduce loop overhead. It partitions the work over the input channels and kernel spatial dimensions
// among the threads in each block. Warp-level reduction is used to sum the partial results.

template <int BLOCK_SIZE>
__global__ void conv_transpose2d_forward_kernel_unrolled(
    const float* __restrict__ input,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int N,
    const int C_in,
    const int H_in,
    const int W_in,
    const int C_out,
    const int H_out,
    const int W_out,
    const int kH,
    const int kW,
    const int sH,
    const int sW,
    const int pH,
    const int pW
) {
    // Each block computes one output element
    int out_idx = blockIdx.x;
    if (out_idx >= N * C_out * H_out * W_out) return;

    int ow = out_idx % W_out;
    int oh = (out_idx / W_out) % H_out;
    int oc = (out_idx / (W_out * H_out)) % C_out;
    int n  = out_idx / (W_out * H_out * C_out);

    float sum = 0.0f;

    // Parallelize over input channels; each thread handles a subset
    for (int ic = threadIdx.x; ic < C_in; ic += BLOCK_SIZE) {
        // Unroll the inner loops over kernel height and width
        #pragma unroll
        for (int kh = 0; kh < kH; kh++) {
            #pragma unroll
            for (int kw = 0; kw < kW; kw++) {
                int i_val = oh + pH - kh;
                int j_val = ow + pW - kw;
                if ((i_val % sH == 0) && (j_val % sW == 0)) {
                    int i_in = i_val / sH;
                    int j_in = j_val / sW;
                    if (i_in >= 0 && i_in < H_in && j_in >= 0 && j_in < W_in) {
                        int input_idx = ((n * C_in + ic) * H_in + i_in) * W_in + j_in;
                        int weight_idx = ((ic * C_out + oc) * kH + kh) * kW + kw;
                        sum += input[input_idx] * c_weight[weight_idx];
                    }
                }
            }
        }
    }

    // Intra-warp reduction using shuffle primitives
    unsigned int mask = 0xffffffff;
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(mask, sum, offset);
    }

    int lane = threadIdx.x & (warpSize - 1);
    __shared__ float shared_sum[32]; // assuming max 32 warps per block
    int warp_id = threadIdx.x / warpSize;
    if (lane == 0) {
        shared_sum[warp_id] = sum;
    }
    __syncthreads();

    // Final reduction across warp sums
    if (threadIdx.x < BLOCK_SIZE / warpSize) {
        float ssum = shared_sum[threadIdx.x];
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            ssum += __shfl_down_sync(mask, ssum, offset);
        }
        if (threadIdx.x == 0) {
            if (bias != nullptr) {
                ssum += bias[oc];
            }
            output[out_idx] = ssum;
        }
    }
}


// Host function to set up and launch the kernel

torch::Tensor conv_transpose2d_forward(
    torch::Tensor x,
    torch::Tensor weight,
    py::object bias_obj,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding
) {
    int weight_size = weight.numel() * sizeof(float);
    const int max_const_size = 64 * 1024;
    if (weight_size > max_const_size) {
        c10::optional<torch::Tensor> bias = c10::nullopt;
        if (!bias_obj.is_none()) {
            bias = bias_obj.cast<torch::Tensor>();
        }
        return at::conv_transpose2d(x, weight, bias, stride, padding);
    }

    hipMemcpyToSymbol(HIP_SYMBOL(c_weight), weight.data_ptr<float>(), weight_size);

    torch::Tensor bias;
    const float* bias_ptr = nullptr;
    if (!bias_obj.is_none()) {
        bias = bias_obj.cast<torch::Tensor>();
        bias_ptr = bias.data_ptr<float>();
    }

    const int N = x.size(0);
    const int C_in = x.size(1);
    const int H_in = x.size(2);
    const int W_in = x.size(3);

    const int C_out = weight.size(1);
    const int kH = weight.size(2);
    const int kW = weight.size(3);
    const int sH = stride[0];
    const int sW = stride[1];
    const int pH = padding[0];
    const int pW = padding[1];

    int H_out = (H_in - 1) * sH - 2 * pH + kH;
    int W_out = (W_in - 1) * sW - 2 * pW + kW;

    auto output = torch::zeros({N, C_out, H_out, W_out}, x.options());

    // Launch one block per output element, each block with BLOCK_SIZE threads
    const int total_output = N * C_out * H_out * W_out;
    constexpr int BLOCK_SIZE = 256;

    conv_transpose2d_forward_kernel_unrolled<BLOCK_SIZE><<<total_output, BLOCK_SIZE>>>(
        x.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        N, C_in, H_in, W_in,
        C_out, H_out, W_out,
        kH, kW, sH, sW, pH, pW
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward, "Conv Transpose 2D forward with loop unrolling",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride"),
          py::arg("padding"));
}
