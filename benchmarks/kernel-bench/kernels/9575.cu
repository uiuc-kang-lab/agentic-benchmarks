#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// Define a maximum kernel size assumed (adjust if necessary)
#define MAX_KERNEL_SIZE 16

// Optimized CUDA kernel for 2D transposed convolution that maps threads efficiently
// across a 3D grid covering batch size, output height, and output width. This aims to
// balance load across multi-dimensional spaces for better parallel utilization.

__global__ void conv_transpose2d_forward_kernel_thread_block_map(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

  int w_out = blockIdx.x * blockDim.x + threadIdx.x;
  int h_out = blockIdx.y * blockDim.y + threadIdx.y;
  int b = blockIdx.z;
  if (h_out >= out_height || w_out >= out_width || b >= batch_size)
    return;

  for (int o = 0; o < out_channels; ++o) {  // Each block computes for a batch output, parallel across channels
    // Precompute base indices for output location
    int base_h = h_out + padding;
    int base_w = w_out + padding;

    // Precompute valid kernel indices for the h dimension
    int valid_p_count = 0;
    int valid_p[MAX_KERNEL_SIZE];        // stores the valid p index
    int h_in_list[MAX_KERNEL_SIZE];        // stores corresponding h_in
    for (int p = 0; p < kernel_size; p++) {
      int p_dilated = p * dilation;
      if (base_h >= p_dilated && ((base_h - p_dilated) % stride) == 0) {
        int h_in = (base_h - p_dilated) / stride;
        if (h_in < in_height) {
          valid_p[valid_p_count] = p;
          h_in_list[valid_p_count] = h_in;
          valid_p_count++;
        }
      }
    }

    // Precompute valid kernel indices for the w dimension
    int valid_q_count = 0;
    int valid_q[MAX_KERNEL_SIZE];        // stores the valid q index
    int w_in_list[MAX_KERNEL_SIZE];        // stores corresponding w_in
    for (int q = 0; q < kernel_size; q++) {
      int q_dilated = q * dilation;
      if (base_w >= q_dilated && ((base_w - q_dilated) % stride) == 0) {
        int w_in = (base_w - q_dilated) / stride;
        if (w_in < in_width) {
          valid_q[valid_q_count] = q;
          w_in_list[valid_q_count] = w_in;
          valid_q_count++;
        }
      }
    }

    // Initialize the output value with the bias for channel o using read-only cache
    float out_val = __ldg(&bias[o]);

    // Iterate over input channels
    for (int c = 0; c < in_channels; ++c) {
      // Loop over precomputed valid p positions
      for (int i = 0; i < valid_p_count; i++) {
        int p = valid_p[i];
        int h_in = h_in_list[i];
        // Loop over precomputed valid q positions
        for (int j = 0; j < valid_q_count; j++) {
          int q = valid_q[j];
          int w_in = w_in_list[j];
          
          // Compute flat indices for input and weight tensors
          int input_idx = (((b * in_channels + c) * in_height) + h_in) * in_width + w_in;
          int weight_idx = (((c * out_channels + o) * kernel_size + p) * kernel_size) + q;
          
          // Accumulate contributions using read-only loads
          out_val += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
        }
      }
    }

    // Write the computed result to the output
    int output_idx = (((b * out_channels) + o) * out_height + h_out) * out_width + w_out;
    output[output_idx] = out_val;
  }
}

// CUDA forward function using efficient 3D thread and block mapping
torch::Tensor conv_transpose2d_forward_cuda_thread_block_map(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {
  // Get dimensions from input and weight tensors
  int batch_size = input.size(0);
  int in_channels = input.size(1);
  int in_height = input.size(2);
  int in_width = input.size(3);

  // Weight tensor has shape: [in_channels, out_channels, kernel_size, kernel_size]
  int out_channels = weight.size(1);
  int kernel_size = weight.size(2);  // assume square kernel

  // Compute output dimensions
  int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  int out_width  = (in_width - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;

  auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());

  dim3 threads(32, 8, 1);  // 32x8 threads per block
  dim3 blocks((out_width + threads.x - 1) / threads.x, (out_height + threads.y - 1) / threads.y, batch_size);

  conv_transpose2d_forward_kernel_thread_block_map<<<blocks, threads>>>(
      input.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      output.data_ptr<float>(),
      batch_size,
      in_channels,
      out_channels,
      in_height,
      in_width,
      kernel_size,
      out_height,
      out_width,
      stride,
      padding,
      dilation);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in conv_transpose2d_forward_kernel_thread_block_map: %s\n", hipGetErrorString(err));
  }

  return output;
}

// Wrapper function to support bias being None (creates a zero bias tensor if needed)
torch::Tensor conv_transpose2d_forward_wrapper_thread_block_map(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,
    int stride,
    int padding,
    int dilation) {
  int out_channels = weight.size(1);
  torch::Tensor bias;
  if (bias_obj.is(pybind11::none())) {
    bias = torch::zeros({out_channels}, weight.options());
  } else {
    bias = bias_obj.cast<torch::Tensor>();
  }
  return conv_transpose2d_forward_cuda_thread_block_map(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &conv_transpose2d_forward_wrapper_thread_block_map,
        "ConvTranspose2d forward (CUDA) with optimal 3D mapping",
        pybind11::arg("input"),
        pybind11::arg("weight"),
        pybind11::arg("bias"),
        pybind11::arg("stride"),
        pybind11::arg("padding"),
        pybind11::arg("dilation"));
}
