#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define TILE_SIZE 16

// Optimized matrix multiplication kernel leveraging shared memory
__global__ void optimized_matmul_kernel(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int M, int K, int N) {
    // Shared memory for tiles of A and B
    __shared__ float shared_A[TILE_SIZE][TILE_SIZE];
    __shared__ float shared_B[TILE_SIZE][TILE_SIZE];

    // Calculate row and column index of C
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    // Loop over tiles
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Load data into shared memory
        if (row < M && (t * TILE_SIZE + threadIdx.x) < K) {
            shared_A[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_SIZE + threadIdx.x];
        } else {
            shared_A[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && (t * TILE_SIZE + threadIdx.y) < K) {
            shared_B[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            shared_B[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        // Multiply the two matrices
        #pragma unroll
        for (int i = 0; i < TILE_SIZE; i++) {
            sum += shared_A[threadIdx.y][i] * shared_B[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Write the result to C if within bounds
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

// Host function to launch the CUDA kernel
torch::Tensor matmul_cuda(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.is_contiguous(), "A must be contiguous");
    TORCH_CHECK(B.is_contiguous(), "B must be contiguous");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());

    // Configure block and grid dimensions
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);

    // Launch the optimized kernel
    optimized_matmul_kernel<<<grid, block>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, K, N);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_cuda, "Optimized matrix multiplication using shared memory (CUDA)");
}
