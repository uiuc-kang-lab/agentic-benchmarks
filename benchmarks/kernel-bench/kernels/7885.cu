#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {
    
    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    if (bias.has_value()) {
        return torch::conv2d(x, weight, bias.value(), {stride, stride}, {padding, padding}, {dilation, dilation}, groups);
    } else {
        return torch::conv2d(x, weight, torch::Tensor(), {stride, stride}, {padding, padding}, {dilation, dilation}, groups);
    }
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA forward function for 2D convolution with optional bias");
}