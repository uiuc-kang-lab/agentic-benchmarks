#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// GELU function for float precision
__device__ inline float gelu_function(float x) {
    return x * 0.5f * (1.0f + erff(x / 1.4142135623730951f));
}

// Vectorized CUDA kernel with overlapped memory transfers via CUDA streams
__global__ void gelu_kernel_vectorized_streams(const float4* __restrict__ input,
                                               float4* __restrict__ output,
                                               size_t n4) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n4) {
        float4 in_val = __ldg(&input[idx]);
        in_val.x = gelu_function(in_val.x);
        in_val.y = gelu_function(in_val.y);
        in_val.z = gelu_function(in_val.z);
        in_val.w = gelu_function(in_val.w);
        output[idx] = in_val;
    }
}

// Kernel for the remaining elements not divisible by float4
__global__ void gelu_kernel_remainder_streams(const float* __restrict__ input,
                                              float* __restrict__ output,
                                              size_t offset,
                                              size_t numel) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t pos = idx + offset;
    if (pos < numel) {
        float val = __ldg(&input[pos]);
        output[pos] = gelu_function(val);
    }
}

// Forward function callable from Python
torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
    TORCH_CHECK(x.scalar_type() == torch::ScalarType::Float,
                "Only float32 is supported for the streamed version");

    auto output = torch::empty_like(x);
    size_t numel = x.numel();
    const size_t vec_size = 4; // float4 => 4 floats (128 bits)
    size_t n4 = numel / vec_size;
    size_t remainder = numel % vec_size;

    int threads = 256;
    int blocks_main = (n4 + threads - 1) / threads;
    int blocks_rem = (remainder + threads - 1) / threads;

    hipStream_t stream_main, stream_rem;
    hipStreamCreate(&stream_main);
    hipStreamCreate(&stream_rem);

    if (n4 > 0) {
        gelu_kernel_vectorized_streams<<<blocks_main, threads, 0, stream_main>>>(
            reinterpret_cast<const float4*>(x.data_ptr<float>()),
            reinterpret_cast<float4*>(output.data_ptr<float>()),
            n4);
    }

    if (remainder > 0) {
        gelu_kernel_remainder_streams<<<blocks_rem, threads, 0, stream_rem>>>(
            x.data_ptr<float>() + n4 * vec_size,
            output.data_ptr<float>() + n4 * vec_size,
            n4 * vec_size,
            numel);
    }

    hipStreamSynchronize(stream_main);
    hipStreamSynchronize(stream_rem);
    hipStreamDestroy(stream_main);
    hipStreamDestroy(stream_rem);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "GELU activation forward (CUDA) with streams");
}
