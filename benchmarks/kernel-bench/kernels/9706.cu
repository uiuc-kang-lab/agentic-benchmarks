#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

// Optimized kernel that leverages __ldg() for read-only global memory accesses and
// assumes that the pointers provided by PyTorch are aligned to 128-bit boundaries.
// This minimizes memory latency by using the read-only data cache for input, weight, and bias.
__global__ void depthwise_conv2d_kernel_aligned_ldg(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch,
    int channels,
    int in_h,
    int in_w,
    int out_h,
    int out_w,
    int kernel_h,
    int stride,
    int padding,
    int dilation) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch * channels * out_h * out_w;
    if (index < total) {
        // Decode the flattened index into 4D coordinates: (b, c, oh, ow)
        int ow = index % out_w;
        int tmp = index / out_w;
        int oh = tmp % out_h;
        tmp = tmp / out_h;
        int c = tmp % channels;
        int b = tmp / channels;

        float sum = 0.f;
        // Unroll the kernel height loop for reduced overhead
        #pragma unroll
        for (int kh = 0; kh < kernel_h; ++kh) {
            int ih = oh * stride - padding + kh * dilation;
            int iw = ow * stride - padding;  // kernel width is 1
            if (ih >= 0 && ih < in_h && iw >= 0 && iw < in_w) {
                int input_idx = ((b * channels + c) * in_h + ih) * in_w + iw;
                int weight_idx = c * kernel_h + kh;  // weight shape: (channels, 1, kernel_h, 1)
                float in_val = __ldg(&input[input_idx]);
                float w_val = __ldg(&weight[weight_idx]);
                sum += in_val * w_val;
            }
        }
        sum += __ldg(&bias[c]);
        int out_idx = ((b * channels + c) * out_h + oh) * out_w + ow;
        output[out_idx] = sum;
    }
}

// Forward function for the depthwise convolution op. It ensures inputs are contiguous and
// sets up the CUDA kernel launch parameters.
at::Tensor forward(
    at::Tensor x,
    at::Tensor weight,
    c10::optional<at::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    // Ensure the inputs are contiguous.
    x = x.contiguous();
    weight = weight.contiguous();

    int batch = x.size(0);
    int channels = x.size(1);
    int in_h = x.size(2);
    int in_w = x.size(3);
    int kernel_h = weight.size(2);  // weight shape: (channels, 1, kernel_h, 1)

    // Depthwise convolution requires groups == channels
    if (groups != channels) {
        throw std::invalid_argument("Depthwise convolution requires groups == number of input channels.");
    }

    // Handle bias: if bias is not provided, use a zeros tensor
    at::Tensor bias_val;
    if (bias.has_value() && bias.value().defined()) {
        bias_val = bias.value().contiguous();
    } else {
        bias_val = at::zeros({channels}, x.options());
    }

    // Compute output dimensions
    int out_h = (in_h + 2 * padding - dilation * (kernel_h - 1) - 1) / stride + 1;
    int out_w = (in_w + 2 * padding - 1) / stride + 1;

    auto output = at::empty({batch, channels, out_h, out_w}, x.options());

    int total = batch * channels * out_h * out_w;
    int threads = 1024;
    int blocks = (total + threads - 1) / threads;

    const float* x_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias_val.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();

    // Launch the optimized kernel
    depthwise_conv2d_kernel_aligned_ldg<<<blocks, threads>>>(
        x_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch,
        channels,
        in_h,
        in_w,
        out_h,
        out_w,
        kernel_h,
        stride,
        padding,
        dilation
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Depthwise 2D Convolution forward with aligned __ldg loads (CUDA)",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = c10::nullopt,
          py::arg("stride"),
          py::arg("padding"),
          py::arg("dilation"),
          py::arg("groups"));
}
