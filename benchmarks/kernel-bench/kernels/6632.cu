#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized kernel using __ldg for read-only memory access and aligning memory accesses

template <typename scalar_t>
__global__ void ldg_max_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t inner_size,
    const int64_t dim_size
) {
    // Shared memory for partial reductions
    extern __shared__ char shared_memory[];
    scalar_t* shared_data = reinterpret_cast<scalar_t*>(shared_memory);
    
    // Determine which outer index this block is working on
    int outer_idx = blockIdx.x;
    int tid = threadIdx.x;
    
    // Determine the tile index in the inner dimension
    int inner_idx = blockIdx.y * blockDim.x + tid;
    if (inner_idx >= inner_size) return;

    // Base offset for this outer index
    int64_t base_offset = outer_idx * dim_size * inner_size;

    // Initialize maximum value with the first element in the reduction dimension
    scalar_t max_val = __ldg(&input[base_offset + inner_idx]);

    // Loop over the reduction dimension
    for (int i = 1; i < dim_size; i++) {
        scalar_t val = __ldg(&input[base_offset + i * inner_size + inner_idx]);
        max_val = max(max_val, val);
    }

    // Store in shared memory
    shared_data[tid] = max_val;
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] = max(shared_data[tid], shared_data[tid + stride]);
        }
        __syncthreads();
    }

    // Write the result to output - only first thread writes result
    if (tid == 0) {
        output[outer_idx * inner_size + blockIdx.y * blockDim.x] = shared_data[0];
    }
}

// Function to launch the optimized kernel
torch::Tensor ldg_max_reduce_cuda_forward(torch::Tensor input, int64_t dim) {
    // Handle negative dimension
    if (dim < 0) dim += input.dim();

    // Calculate sizes
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= input.size(i);
    }
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < input.dim(); i++) {
        inner_size *= input.size(i);
    }
    
    const int64_t dim_size = input.size(dim);

    // Create output tensor
    auto output_sizes = input.sizes().vec();
    output_sizes.erase(output_sizes.begin() + dim);
    auto output = torch::empty(output_sizes, input.options());

    // Configure block and grid sizes
    const int threads = 256;
    int blocks_y = (inner_size + threads - 1) / threads;
    dim3 grid(outer_size, blocks_y);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "ldg_max_reduce_forward", ([&] {
        ldg_max_reduce_kernel<scalar_t><<<grid, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            inner_size,
            dim_size
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &ldg_max_reduce_cuda_forward, "LDG Max reduction forward (CUDA)");
}