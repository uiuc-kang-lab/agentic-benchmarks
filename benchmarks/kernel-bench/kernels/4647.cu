#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void rms_norm_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int num_features,
    const int numel_per_batch,
    const float eps
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int total_elements = batch_size * numel_per_batch;
    
    __shared__ scalar_t shared_sumsq[256];
    
    for (int idx = tid; idx < total_elements; idx += stride) {
        const int batch_id = idx / numel_per_batch;
        const int offset_in_batch = idx % numel_per_batch;
        const int batch_offset = batch_id * num_features * numel_per_batch;
        
        scalar_t sumsq = 0.0f;
        int feat = 0;
        
        #pragma unroll 8
        for (; feat < (num_features / 8) * 8; feat += 8) {
            #pragma unroll
            for (int i = 0; i < 8; i++) {
                const scalar_t val = input[batch_offset + (feat + i) * numel_per_batch + offset_in_batch];
                sumsq += val * val;
            }
        }
        
        for (; feat < num_features; feat++) {
            const scalar_t val = input[batch_offset + feat * numel_per_batch + offset_in_batch];
            sumsq += val * val;
        }
        
        const scalar_t rms = sqrt(sumsq / num_features + eps);
        
        feat = 0;
        #pragma unroll 8
        for (; feat < (num_features / 8) * 8; feat += 8) {
            #pragma unroll
            for (int i = 0; i < 8; i++) {
                const int out_idx = batch_offset + (feat + i) * numel_per_batch + offset_in_batch;
                output[out_idx] = input[out_idx] / rms;
            }
        }
        
        for (; feat < num_features; feat++) {
            const int out_idx = batch_offset + feat * numel_per_batch + offset_in_batch;
            output[out_idx] = input[out_idx] / rms;
        }
    }
}

torch::Tensor rms_norm_cuda_forward(torch::Tensor input, float eps) {
    auto output = torch::empty_like(input);
    
    const int batch_size = input.size(0);
    const int num_features = input.size(1);
    
    int numel_per_batch = 1;
    for(int i = 2; i < input.dim(); i++) {
        numel_per_batch *= input.size(i);
    }

    const int threads_per_block = 256;
    const int total_elements = batch_size * numel_per_batch;
    
    int device_id;
    hipGetDevice(&device_id);
    int sm_count;
    hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device_id);
    const int blocks = min(65535, sm_count * 32);

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "rms_norm_cuda", ([&] {
        rms_norm_kernel<scalar_t><<<blocks, threads_per_block, 0, stream>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            num_features,
            numel_per_batch,
            eps
        );
    }));

    return output;
}