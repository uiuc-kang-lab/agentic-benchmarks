#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel that flattens the entire output index space and uses a grid-stride loop
// to evenly distribute work across threads and blocks.
__global__ void avg_pool3d_forward_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int batch_size, int channels,
    int in_d, int in_h, int in_w,
    int out_d, int out_h, int out_w,
    int kernel_size, int stride, int padding) {

    int total_elements = batch_size * channels * out_d * out_h * out_w;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride_total = blockDim.x * gridDim.x;

    while (tid < total_elements) {
        // Decompose linear index into (n, c, d_out, h_out, w_out)
        int tmp = tid;
        int w_out = tmp % out_w;
        tmp /= out_w;
        int h_out = tmp % out_h;
        tmp /= out_h;
        int d_out = tmp % out_d;
        tmp /= out_d;
        int c = tmp % channels;
        int n = tmp / channels;

        // Compute pooling window boundaries
        int d_start = d_out * stride - padding;
        int h_start = h_out * stride - padding;
        int w_start = w_out * stride - padding;
        int d_end = d_start + kernel_size;
        int h_end = h_start + kernel_size;
        int w_end = w_start + kernel_size;

        // Clamp pooling window boundaries to input dimensions
        int d_start_clamped = d_start < 0 ? 0 : d_start;
        int h_start_clamped = h_start < 0 ? 0 : h_start;
        int w_start_clamped = w_start < 0 ? 0 : w_start;
        int d_end_clamped = d_end > in_d ? in_d : d_end;
        int h_end_clamped = h_end > in_h ? in_h : h_end;
        int w_end_clamped = w_end > in_w ? in_w : w_end;

        float sum = 0.0f;
        // Iterate over the pooling window
        for (int d = d_start_clamped; d < d_end_clamped; d++) {
            for (int h = h_start_clamped; h < h_end_clamped; h++) {
                // Compute starting index for the row in input
                int row_base = (((n * channels + c) * in_d + d) * in_h + h) * in_w;
                for (int w = w_start_clamped; w < w_end_clamped; w++) {
                    sum += input[row_base + w];
                }
            }
        }

        // Divide by full pooling volume (count_include_pad=True)
        int pool_volume = kernel_size * kernel_size * kernel_size;

    // Cache pool_volume for all iterations
    
    while (tid < total_elements) {
        // Decompose linear index into (n, c, d_out, h_out, w_out)
        int tmp = tid;
        int w_out = tmp % out_w;
        tmp /= out_w;
        int h_out = tmp % out_h;
        tmp /= out_h;
        int d_out = tmp % out_d;
        tmp /= out_d;
        int c = tmp % channels;
        int n = tmp / channels;

        // Compute pooling window boundaries
        int d_start = d_out * stride - padding;
        int h_start = h_out * stride - padding;
        int w_start = w_out * stride - padding;
        int d_end = d_start + kernel_size;
        int h_end = h_start + kernel_size;
        int w_end = w_start + kernel_size;

        // Clamp pooling window boundaries to input dimensions
        int d_start_clamped = d_start < 0 ? 0 : d_start;
        int h_start_clamped = h_start < 0 ? 0 : h_start;
        int w_start_clamped = w_start < 0 ? 0 : w_start;
        int d_end_clamped = d_end > in_d ? in_d : d_end;
        int h_end_clamped = h_end > in_h ? in_h : h_end;
        int w_end_clamped = w_end > in_w ? in_w : w_end;

        float sum = 0.0f;

        // Precompute constant term for input offset
        int n_ch = n * channels + c;

        // Iterate over the pooling window with cached multiplications
        for (int d = d_start_clamped; d < d_end_clamped; d++) {
            int base_d = (n_ch * in_d + d) * in_h * in_w;
            for (int h = h_start_clamped; h < h_end_clamped; h++) {
                int base_row = base_d + h * in_w;
                for (int w = w_start_clamped; w < w_end_clamped; w++) {
                    sum += input[base_row + w];
                }
            }
        }

        // Divide by full pooling volume (count_include_pad=True)
        output[tid] = sum / static_cast<float>(pool_volume);

        tid += stride_total;
    }
    }
}

at::Tensor forward(at::Tensor input, int kernel_size, int stride, int padding) {
    TORCH_CHECK(input.dim() == 5, "Input tensor must be 5-dimensional");
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    int batch_size = input.size(0);
    int channels   = input.size(1);
    int in_d       = input.size(2);
    int in_h       = input.size(3);
    int in_w       = input.size(4);

    // Compute output dimensions
    int out_d = (in_d + 2 * padding - kernel_size) / stride + 1;
    int out_h = (in_h + 2 * padding - kernel_size) / stride + 1;
    int out_w = (in_w + 2 * padding - kernel_size) / stride + 1;

    auto output = at::empty({batch_size, channels, out_d, out_h, out_w}, input.options());
    int total_elements = batch_size * channels * out_d * out_h * out_w;

    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;

    avg_pool3d_forward_kernel<<<blocks, threads>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size, channels,
        in_d, in_h, in_w,
        out_d, out_h, out_w,
        kernel_size, stride, padding);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D Average Pooling forward (CUDA) with balanced workload distribution");
}
