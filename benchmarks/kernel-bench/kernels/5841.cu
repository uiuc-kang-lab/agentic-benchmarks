#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>
#include <cmath>

// Inline device function to compute minimum of two integers
__device__ inline int dmin(int a, int b) {
    return a < b ? a : b;
}

// Combined CUDA kernel for 3D max pooling that minimizes warp divergence
// by precomputing valid loop bounds and unrolling inner loops

template <typename scalar_t>
__global__ void max_pool3d_forward_combined_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t* __restrict__ indices,
    const int batch_size,
    const int channels,
    const int input_d, const int input_h, const int input_w,
    const int output_d, const int output_h, const int output_w,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation) {

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total = batch_size * channels * output_d * output_h * output_w;
    if (idx >= total) return;

    // Compute output indices
    const int w_out = idx % output_w;
    const int h_out = (idx / output_w) % output_h;
    const int d_out = (idx / (output_w * output_h)) % output_d;
    const int c = (idx / (output_w * output_h * output_d)) % channels;
    const int b = idx / (output_w * output_h * output_d * channels);

    // Compute starting positions in input tensor
    const int d_start = d_out * stride - padding;
    const int h_start = h_out * stride - padding;
    const int w_start = w_out * stride - padding;

    // Precompute valid loop bounds for each dimension to avoid branch divergence
    const int k_d_min = (d_start < 0) ? ((-d_start + dilation - 1) / dilation) : 0;
    const int valid_d_max = (input_d - d_start + dilation - 1) / dilation;
    const int k_d_max = dmin(kernel_size, valid_d_max);

    const int k_h_min = (h_start < 0) ? ((-h_start + dilation - 1) / dilation) : 0;
    const int valid_h_max = (input_h - h_start + dilation - 1) / dilation;
    const int k_h_max = dmin(kernel_size, valid_h_max);

    const int k_w_min = (w_start < 0) ? ((-w_start + dilation - 1) / dilation) : 0;
    const int valid_w_max = (input_w - w_start + dilation - 1) / dilation;
    const int k_w_max = dmin(kernel_size, valid_w_max);

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    int max_index = -1;

    // Loop over the valid pooling window with unrolling to reduce loop overhead
    #pragma unroll
    for (int kd = k_d_min; kd < k_d_max; kd++) {
        const int d_in = d_start + kd * dilation;
        #pragma unroll
        for (int kh = k_h_min; kh < k_h_max; kh++) {
            const int h_in = h_start + kh * dilation;
            #pragma unroll
            for (int kw = k_w_min; kw < k_w_max; kw++) {
                const int w_in = w_start + kw * dilation;
                const int input_idx = (((b * channels + c) * input_d + d_in) * input_h + h_in) * input_w + w_in;
                const scalar_t val = input[input_idx];
                if (val > max_val) {
                    max_val = val;
                    max_index = input_idx;
                }
            }
        }
    }

    output[idx] = max_val;
    if (indices != nullptr) {
        indices[idx] = max_index;
    }
}

// Host function that sets up the kernel launch and computes output dimensions

torch::Tensor max_pool3d_cuda_forward_combined(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation,
    bool return_indices,
    bool ceil_mode) {

    const auto input_sizes = input.sizes();
    const int batch_size = input_sizes[0];
    const int channels = input_sizes[1];
    const int input_d = input_sizes[2];
    const int input_h = input_sizes[3];
    const int input_w = input_sizes[4];

    // Calculate output dimensions based on ceil_mode
    int output_d = ceil_mode ?
        std::ceil((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        std::floor((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    int output_h = ceil_mode ?
        std::ceil((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        std::floor((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    int output_w = ceil_mode ?
        std::ceil((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        std::floor((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);

    auto output = torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options());
    auto indices = return_indices ?
        torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options().dtype(torch::kLong)) :
        torch::Tensor();

    const int threads = 256;
    const int total = batch_size * channels * output_d * output_h * output_w;
    const int blocks = (total + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool3d_forward_cuda_combined", ([&] {
        max_pool3d_forward_combined_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            return_indices ? indices.data_ptr<int64_t>() : nullptr,
            batch_size, channels,
            input_d, input_h, input_w,
            output_d, output_h, output_w,
            kernel_size, stride, padding, dilation);
    }));

    if (return_indices) {
        return torch::stack({output, indices}, 0);
    }
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool3d_cuda_forward_combined, "Max Pool 3D forward combined optimized (CUDA)");
}
