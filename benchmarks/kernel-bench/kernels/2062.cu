#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef TILE_SIZE
#define TILE_SIZE 32
#endif

__global__ void triangular_mm_kernel_vectorized(const float* __restrict__ A,
                                              const float* __restrict__ B,
                                              float* __restrict__ C,
                                              const int N) {
    const int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    const int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    if (row < N && col < N) {
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            float sum = 0.f;
            
            // Main loop with manual unrolling by 4
            const int end_aligned = col + ((row - col + 1) & ~3);
            int k = col;
            
            #pragma unroll
            for (; k < end_aligned - 3; k += 4) {
                sum += A[row * N + k] * B[k * N + col];
                sum += A[row * N + (k+1)] * B[(k+1) * N + col];
                sum += A[row * N + (k+2)] * B[(k+2) * N + col];
                sum += A[row * N + (k+3)] * B[(k+3) * N + col];
            }
            
            // Handle remaining elements
            #pragma unroll
            for (; k <= row; k++) {
                sum += A[row * N + k] * B[k * N + col];
            }
            
            C[row * N + col] = sum;
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    triangular_mm_kernel_vectorized<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Unrolled and vectorized triangular matrix multiplication (CUDA)");
}