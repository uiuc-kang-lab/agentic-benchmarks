#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void mean_reduce_kernel(
    const scalar_t* input,
    scalar_t* output,
    int64_t outer_size,
    int64_t dim_size,
    int64_t inner_size) {
    
    extern __shared__ char shared_mem[];
    scalar_t* shared_data = reinterpret_cast<scalar_t*>(shared_mem);
    
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int output_idx = bid;
    
    if (output_idx >= outer_size * inner_size) return;
    
    const int outer_idx = output_idx / inner_size;
    const int inner_idx = output_idx % inner_size;
    const int input_offset = outer_idx * dim_size * inner_size + inner_idx;
    
    // Initialize shared memory with zeros
    shared_data[tid] = 0;
    
    // Each thread loads and adds multiple elements
    scalar_t thread_sum = 0;
    for (int i = tid; i < dim_size; i += blockDim.x) {
        thread_sum += input[input_offset + i * inner_size];
    }
    shared_data[tid] = thread_sum;
    __syncthreads();
    
    // Parallel reduction in shared memory
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }
    
    // Write result
    if (tid == 0) {
        output[output_idx] = shared_data[0] / dim_size;
    }
}

torch::Tensor mean_reduce_cuda(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();
    
    auto sizes = input.sizes().vec();
    int64_t dim_size = sizes[dim];
    
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }
    
    sizes.erase(sizes.begin() + dim);
    auto output = torch::empty(sizes, input.options());
    
    const int threads = 256;
    const int blocks = outer_size * inner_size;
    const int shared_mem_size = threads * sizeof(float);
    
    AT_DISPATCH_FLOATING_TYPES(input.type(), "mean_reduce_cuda", ([&] {
        mean_reduce_kernel<scalar_t><<<blocks, threads, shared_mem_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            dim_size,
            inner_size
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &mean_reduce_cuda, "Mean reduction (CUDA)");
}