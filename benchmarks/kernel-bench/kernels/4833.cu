#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

#define WARP_SIZE 32

// CUDA kernel implementing L1 normalization with loop unrolling, __ldg(), and vectorized accesses
__global__ void l1_norm_forward_kernel_unrolled(const float* __restrict__ x,
                                                float* __restrict__ out,
                                                int N,
                                                int D) {
  int row = blockIdx.x;
  float thread_sum = 0.0f;

  // Check if vectorized (128-bit) loads can be used
  bool vec4_possible = (D % 4 == 0);

  if (vec4_possible) {
    int vec_count = D / 4;
    const float4* x_vec = reinterpret_cast<const float4*>(x);
    // Unroll loop by 4
    #pragma unroll 4
    for (int i = threadIdx.x; i < vec_count; i += blockDim.x) {
      float4 data = __ldg(&x_vec[row * vec_count + i]);
      thread_sum += fabsf(data.x) + fabsf(data.y) + fabsf(data.z) + fabsf(data.w);
    }
  } else {
    // Fallback to scalar loads with __ldg for non 128-bit aligned cases
    #pragma unroll 4
    for (int col = threadIdx.x; col < D; col += blockDim.x) {
      float data = __ldg(&x[row * D + col]);
      thread_sum += fabsf(data);
    }
  }

  // Warp-level reduction using shuffle intrinsics
  unsigned int lane = threadIdx.x % WARP_SIZE;
  for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
    thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
  }

  // Shared memory reduction across warps
  extern __shared__ float sdata[];
  int warp_id = threadIdx.x / WARP_SIZE;
  if (lane == 0) {
    sdata[warp_id] = thread_sum;
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    float total_sum = 0.0f;
    int num_warps = (blockDim.x + WARP_SIZE - 1) / WARP_SIZE;
    for (int i = 0; i < num_warps; i++) {
      total_sum += sdata[i];
    }
    if (total_sum == 0.0f) {
      total_sum = 1e-12f;
    }
    sdata[0] = total_sum;
  }
  __syncthreads();
  float total_sum = sdata[0];

  // Normalize the row elements
  if (vec4_possible) {
    int vec_count = D / 4;
    const float4* x_vec = reinterpret_cast<const float4*>(x);
    float4* out_vec = reinterpret_cast<float4*>(out);
    // Unroll loop by 4
    #pragma unroll 4
    for (int i = threadIdx.x; i < vec_count; i += blockDim.x) {
      float4 data = __ldg(&x_vec[row * vec_count + i]);
      data.x = data.x / total_sum;
      data.y = data.y / total_sum;
      data.z = data.z / total_sum;
      data.w = data.w / total_sum;
      out_vec[row * vec_count + i] = data;
    }
  } else {
    // Unroll loop by 4
    #pragma unroll 4
    for (int col = threadIdx.x; col < D; col += blockDim.x) {
      float data = __ldg(&x[row * D + col]);
      out[row * D + col] = data / total_sum;
    }
  }
}

// Host function to launch the CUDA kernel

torch::Tensor forward(torch::Tensor x) {
  TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA.");
  TORCH_CHECK(x.dim() == 2, "Expected 2D tensor.");
  x = x.contiguous();

  auto out = torch::empty_like(x);
  int N = x.size(0);
  int D = x.size(1);

  // Determine the number of threads per block (up to 1024, but not more than D)
  int threads = (D < 1024) ? D : 1024;
  int num_warps = (threads + WARP_SIZE - 1) / WARP_SIZE;
  int shared_mem_size = num_warps * sizeof(float);

  l1_norm_forward_kernel_unrolled<<<N, threads, shared_mem_size>>>(
    x.data_ptr<float>(),
    out.data_ptr<float>(),
    N,
    D
  );

  return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "L1 Normalization forward pass with loop unrolling (CUDA)");
}