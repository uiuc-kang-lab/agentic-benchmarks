#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    int bx = blockIdx.x * TILE_SIZE;
    int by = blockIdx.y * TILE_SIZE;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by + ty;
    int col = bx + tx;
    
    float sum = 0.0f;
    
    // Only compute for lower triangular portion
    if (row >= col && row < N && col < N) {
        // Loop over tiles
        for (int t = col; t <= row; t += TILE_SIZE) {
            // Load tile from A - only load if within bounds and lower triangular
            if (t + tx <= row && row < N) {
                As[ty][tx] = A[row * N + (t + tx)];
            } else {
                As[ty][tx] = 0.0f;
            }
            
            // Load tile from B - only load if within bounds and lower triangular
            if (col < N && t + ty >= col) {
                Bs[ty][tx] = B[(t + ty) * N + col];
            } else {
                Bs[ty][tx] = 0.0f;
            }
            
            // Ensure all threads have loaded their data
            __syncthreads();
            
            // Compute partial dot product for this tile
            for (int k = 0; k < TILE_SIZE; k++) {
                if (t + k <= row) {
                    sum += As[ty][k] * Bs[k][tx];
                }
            }
            
            // Ensure computation is complete before loading next tile
            __syncthreads();
        }
        
        // Write result
        C[row * N + col] = sum;
    } else if (row < col && row < N && col < N) {
        // Upper triangular portion is zero
        C[row * N + col] = 0.0f;
    }
}

// C++ interface exposed to PyTorch
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE,
                   (N + TILE_SIZE - 1) / TILE_SIZE);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}