#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

__global__ void reverse_cumsum_kernel(const float* __restrict__ x, float* __restrict__ out, int64_t size, int64_t /*dim_stride*/) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Compute the reversed index: we want out[r_idx] = sum(x[r_idx:])
        int r_idx = size - 1 - idx;
        float sum = 0.0f;
        // Perform the cumulative sum from r_idx to the end
        for (int j = r_idx; j < size; ++j) {
            sum += x[j];
        }
        out[r_idx] = sum;
    }
}

at::Tensor reverse_cumsum_optimized(at::Tensor x, int64_t dim) {
    // Ensure the tensor is contiguous and on CUDA
    x = x.contiguous();
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");

    at::Tensor out = at::zeros_like(x);
    int64_t size = x.numel();
    int threads = 256;
    int blocks = (size + threads - 1) / threads;

    // Calculate strides for dimension
    int64_t dim_stride = 1;
    for (int i = 0; i < dim; ++i) {
        dim_stride *= x.size(i);
    }

    // Launch kernel
    reverse_cumsum_kernel<<<blocks, threads, threads * sizeof(float)>>>(
        x.data_ptr<float>(),
        out.data_ptr<float>(),
        size,
        dim_stride
    );

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &reverse_cumsum_optimized, "Reverse cumulative sum with memory coalescing (CUDA)");
}