#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void transpose_kernel(const float* __restrict__ in, float* __restrict__ out, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width && y < height) {
        out[y + x * height] = __ldg(&in[x + y * width]);
    }
}

void transpose(const float* in, float* out, int width, int height, hipStream_t stream) {
    dim3 threads(16, 16);
    dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
    transpose_kernel<<<blocks, threads, 0, stream>>>(in, out, width, height);
}

torch::Tensor matmul_cuda(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::empty({M, N}, A.options());

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0;
    const float beta = 0.0;

    // Use a separate buffer for B and A to hold their transposes
    torch::Tensor B_t = torch::empty({K, N}, A.options());
    torch::Tensor A_t = torch::empty({M, K}, A.options());

    // Transpose A and B, then perform GEMM
    transpose(A.data_ptr<float>(), A_t.data_ptr<float>(), K, M, nullptr);
    transpose(B.data_ptr<float>(), B_t.data_ptr<float>(), N, K, nullptr);

    hipblasSgemm(handle,
                HIPBLAS_OP_N,   // Use B as is (but treat as column-major after transpose)
                HIPBLAS_OP_N,   // Use A as is (but treat as column-major after transpose)
                N, M, K,       // Dimensions are swapped because we're computing (A*B)^T
                &alpha,
                B_t.data_ptr<float>(), N,  // Leading dimension is N for B_t
                A_t.data_ptr<float>(), K,  // Leading dimension is K for A_t
                &beta,
                C.data_ptr<float>(), N);   // Leading dimension is N for C

    hipblasDestroy(handle);
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_cuda, "cuBLAS Matrix Multiplication with memory access optimization (CUDA)");
}
