#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")

__constant__ float const_weight[1024];
__constant__ float const_bias[1024];

__global__ void batch_norm_kernel(
    const float* __restrict__ input,
    float* __restrict__ running_mean,
    float* __restrict__ running_var,
    bool training,
    float momentum,
    float eps,
    float* __restrict__ output,
    int N,
    int C,
    int H,
    int W) {
    
    const int c = blockIdx.x;
    const int num_elements = N * H * W;
    const int tid = threadIdx.x;
    const int stride = blockDim.x;

    // Shared memory layout:
    // [input_tile][sum_shared][sum_sq_shared]
    extern __shared__ float smem[];
    const int TILE_SIZE = 128;  // Adjust based on shared memory constraints
    float* input_tile = smem;
    float* sum_shared = &smem[TILE_SIZE];
    float* sum_sq_shared = &smem[TILE_SIZE + blockDim.x];

    float mean, var;
    
    if (training) {
        float sum = 0.0f, sum_sq = 0.0f;
        
        // Process data in tiles
        for (int tile_start = 0; tile_start < num_elements; tile_start += TILE_SIZE) {
            const int tile_elements = min(TILE_SIZE, num_elements - tile_start);
            
            // Load tile into shared memory
            for (int i = tid; i < tile_elements; i += stride) {
                const int global_idx = tile_start + i;
                const int n = global_idx / (H * W);
                const int hw = global_idx % (H * W);
                const int h = hw / W;
                const int w = hw % W;
                const int idx = ((n * C + c) * H + h) * W + w;
                input_tile[i] = input[idx];
            }
            __syncthreads();
            
            // Process tile data
            for (int i = tid; i < tile_elements; i += stride) {
                const float val = input_tile[i];
                sum += val;
                sum_sq += val * val;
            }
            __syncthreads();
        }

        sum_shared[tid] = sum;
        sum_sq_shared[tid] = sum_sq;
        __syncthreads();

        // Block reduction
        for (int s = blockDim.x/2; s > 0; s >>= 1) {
            if (tid < s) {
                sum_shared[tid] += sum_shared[tid + s];
                sum_sq_shared[tid] += sum_sq_shared[tid + s];
            }
            __syncthreads();
        }

        if (tid == 0) {
            const float total_sum = sum_shared[0];
            const float total_sum_sq = sum_sq_shared[0];
            mean = total_sum / num_elements;
            var = (total_sum_sq / num_elements) - (mean * mean);
            
            // Update running statistics
            running_mean[c] = (1 - momentum) * running_mean[c] + momentum * mean;
            running_var[c] = (1 - momentum) * running_var[c] + momentum * var;
            
            // Store in shared mem for next phase
            input_tile[0] = mean;
            input_tile[1] = var;
        }
        __syncthreads();
        
        mean = input_tile[0];
        var = input_tile[1];
    } else {
        mean = running_mean[c];
        var = running_var[c];
    }

    // Phase 2: Normalize and write output using tiling
    const float inv_std = rsqrtf(var + eps);
    const float w = const_weight[c];
    const float b = const_bias[c];

    for (int tile_start = 0; tile_start < num_elements; tile_start += TILE_SIZE) {
        const int tile_elements = min(TILE_SIZE, num_elements - tile_start);
        
        // Load tile into shared memory
        for (int i = tid; i < tile_elements; i += stride) {
            const int global_idx = tile_start + i;
            const int n = global_idx / (H * W);
            const int hw = global_idx % (H * W);
            const int h = hw / W;
            const int w_idx = hw % W;
            const int idx = ((n * C + c) * H + h) * W + w_idx;
            input_tile[i] = input[idx];
        }
        __syncthreads();
        
        // Process and write back
        for (int i = tid; i < tile_elements; i += stride) {
            const int global_idx = tile_start + i;
            const int n = global_idx / (H * W);
            const int hw = global_idx % (H * W);
            const int h = hw / W;
            const int w_idx = hw % W;
            const int idx = ((n * C + c) * H + h) * W + w_idx;
            const float val = input_tile[i];
            output[idx] = (val - mean) * inv_std * w + b;
        }
        __syncthreads();
    }
}

void set_constants(const torch::Tensor& weight, const torch::Tensor& bias) {
    hipMemcpyToSymbol(HIP_SYMBOL(const_weight), weight.data_ptr<float>(), weight.numel() * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(const_bias), bias.data_ptr<float>(), bias.numel() * sizeof(float));
}

torch::Tensor forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    torch::Tensor running_mean,
    torch::Tensor running_var,
    bool training,
    float momentum,
    float eps) {
    
    // Input checks
    CHECK_CUDA(input);
    CHECK_CUDA(weight);
    CHECK_CUDA(bias);
    CHECK_CUDA(running_mean);
    CHECK_CUDA(running_var);
    
    CHECK_CONTIGUOUS(input);
    CHECK_CONTIGUOUS(weight);
    CHECK_CONTIGUOUS(bias);
    CHECK_CONTIGUOUS(running_mean);
    CHECK_CONTIGUOUS(running_var);

    const int N = input.size(0);
    const int C = input.size(1);
    const int H = input.size(2);
    const int W = input.size(3);

    auto output = torch::empty_like(input);
    
    const int threads = 256;
    const size_t shared_mem = 2 * threads * sizeof(float);
    
    set_constants(weight, bias);

    batch_norm_kernel<<<C, threads, shared_mem>>>(
        input.data_ptr<float>(),
        running_mean.data_ptr<float>(),
        running_var.data_ptr<float>(),
        training,
        momentum,
        eps,
        output.data_ptr<float>(),
        N, C, H, W
    );
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_cuda, "BatchNorm forward (CUDA)");
}