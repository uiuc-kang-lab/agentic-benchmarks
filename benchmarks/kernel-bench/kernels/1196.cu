#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <vector>

__global__ void einsum_kernel_streamed(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int BATCH_chunk, int I, int J, int L, int K
) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = BATCH_chunk * I * J * K;
    if (global_idx >= total) return;

    int k = global_idx % K;
    int remainder = global_idx / K;
    int j = remainder % J;
    remainder /= J;
    int i = remainder % I;
    int b_local = remainder / I;

    float sum = 0.0f;
    for(int l = 0; l < L; ++l) {
        int a_offset = b_local * I*J*L + i*J*L + j*L + l;
        int b_offset = l*K + k;
        sum += A[a_offset] * B[b_offset];
    }
    C[global_idx] = sum;
}

// Forward function to launch the kernel with memory overlap
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(A.dim() == 4, "A must be 4D");
    TORCH_CHECK(B.dim() == 2, "B must be 2D");
    TORCH_CHECK(A.size(3) == B.size(0), "Dimension mismatch in l");

    int BATCH = A.size(0);
    int I = A.size(1);
    int J = A.size(2);
    int L = A.size(3);
    int K = B.size(1);

    auto C = torch::zeros({BATCH, I, J, K}, A.options());

    int num_streams = 2;
    int batch_chunk = (BATCH + num_streams - 1) / num_streams;

    std::vector<hipStream_t> streams(num_streams);
    for (int s = 0; s < num_streams; s++) {
        hipStreamCreate(&streams[s]);
    }

    for (int s = 0; s < num_streams; s++) {
        int start_batch = s * batch_chunk;
        int current_batch = std::min(batch_chunk, BATCH - start_batch);
        if (current_batch <= 0) break;

        const float* A_ptr = A.data_ptr<float>() + start_batch * I * J * L;
        float* C_ptr = C.data_ptr<float>() + start_batch * I * J * K;
        int total_elements = current_batch * I * J * K;

        int threads = 256;
        int blocks = (total_elements + threads - 1) / threads;

        einsum_kernel_streamed<<<blocks, threads, 0, streams[s]>>>(
            A_ptr, B.data_ptr<float>(), C_ptr,
            current_batch, I, J, L, K
        );
    }

    for (int s = 0; s < num_streams; s++) {
        hipStreamSynchronize(streams[s]);
        hipStreamDestroy(streams[s]);
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "4D tensor-matrix multiplication with streamed memory overlap (CUDA)");
}
