#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Kernel that uses shared memory for intra-block reduction. Each block handles one batch sample.

template <typename scalar_t>
__global__ void shared_triplet_margin_loss_kernel(
    const scalar_t* __restrict__ anchor,
    const scalar_t* __restrict__ positive,
    const scalar_t* __restrict__ negative,
    scalar_t* __restrict__ output,
    const float margin,
    const int feat_size) {

    // Each block processes one sample in the batch
    const int batch_idx = blockIdx.x;
    const int offset = batch_idx * feat_size;

    // Each thread computes a partial sum over its assigned feature indices
    scalar_t sum_pos = static_cast<scalar_t>(0);
    scalar_t sum_neg = static_cast<scalar_t>(0);

    for (int i = threadIdx.x; i < feat_size; i += blockDim.x) {
        const scalar_t a = anchor[offset + i];
        const scalar_t p = positive[offset + i];
        const scalar_t n = negative[offset + i];
        scalar_t diff_pos = a - p;
        scalar_t diff_neg = a - n;
        sum_pos += diff_pos * diff_pos;
        sum_neg += diff_neg * diff_neg;
    }

    // Allocate shared memory for reduction; we need room for two arrays of size blockDim.x
    extern __shared__ char shared_mem[];
    scalar_t* sdata_pos = reinterpret_cast<scalar_t*>(shared_mem);
    scalar_t* sdata_neg = sdata_pos + blockDim.x;

    sdata_pos[threadIdx.x] = sum_pos;
    sdata_neg[threadIdx.x] = sum_neg;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sdata_pos[threadIdx.x] += sdata_pos[threadIdx.x + s];
            sdata_neg[threadIdx.x] += sdata_neg[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Thread 0 computes the final loss value for the sample
    if (threadIdx.x == 0) {
        scalar_t sqrt_pos = sqrt(sdata_pos[0]);
        scalar_t sqrt_neg = sqrt(sdata_neg[0]);
        scalar_t loss = max(static_cast<scalar_t>(0), sqrt_pos - sqrt_neg + margin);
        output[batch_idx] = loss;
    }
}


// Host function that sets up and launches the kernel
torch::Tensor shared_triplet_margin_loss_cuda(
    torch::Tensor anchor,
    torch::Tensor positive,
    torch::Tensor negative,
    float margin) {

    TORCH_CHECK(anchor.device().is_cuda(), "anchor must be a CUDA tensor");
    TORCH_CHECK(positive.device().is_cuda(), "positive must be a CUDA tensor");
    TORCH_CHECK(negative.device().is_cuda(), "negative must be a CUDA tensor");

    const int batch_size = anchor.size(0);
    const int feat_size = anchor.size(1);

    // Allocate output tensor (one loss per sample)
    auto output = torch::zeros({batch_size}, anchor.options());

    // Launch configuration: one block per sample in the batch
    const int threads = 256;
    dim3 blocks(batch_size);
    
    // Compute shared memory size needed: two arrays, each of size 'threads'
    AT_DISPATCH_FLOATING_TYPES(anchor.scalar_type(), "shared_triplet_margin_loss_kernel", ([&] {
        const int shared_mem_bytes = 2 * threads * sizeof(scalar_t);
        shared_triplet_margin_loss_kernel<scalar_t><<<blocks, threads, shared_mem_bytes>>>(
            anchor.data_ptr<scalar_t>(),
            positive.data_ptr<scalar_t>(),
            negative.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            margin,
            feat_size);
    }));

    return output.mean();
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &shared_triplet_margin_loss_cuda, "Triplet margin loss forward with shared memory (CUDA)");
}
