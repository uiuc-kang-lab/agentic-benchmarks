#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>

template <typename scalar_t>
__device__ __forceinline__ scalar_t warp_reduce_max(scalar_t val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = max(val, __shfl_down_sync(0xffffffff, val, offset));
    }
    return val;
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t warp_reduce_sum(scalar_t val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

template <typename scalar_t, int VEC_SIZE>
__global__ void log_softmax_forward_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int dim_size) {

    const int batch_idx = blockIdx.x;
    const int tid = threadIdx.x;
    const int lane_id = tid % 32;
    const int warp_id = tid / 32;
    const int warps_per_block = blockDim.x / 32;
    
    const scalar_t* __restrict__ input_row = input + batch_idx * dim_size;
    scalar_t* output_row = output + batch_idx * dim_size;

    extern __shared__ __align__(sizeof(scalar_t)) unsigned char smem[];
    scalar_t* warp_max = reinterpret_cast<scalar_t*>(smem);
    scalar_t* warp_sum = warp_max + warps_per_block;

    // Vectorized max reduction
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    const int vec_dim = dim_size / VEC_SIZE;
    for (int idx = tid; idx < vec_dim; idx += blockDim.x) {
        scalar_t vals[VEC_SIZE];
        *reinterpret_cast<typename std::conditional<VEC_SIZE == 4, float4, double2>::type*>(vals) = 
            *reinterpret_cast<const typename std::conditional<VEC_SIZE == 4, float4, double2>::type*>(input_row + idx * VEC_SIZE);
        
        #pragma unroll
        for (int i = 0; i < VEC_SIZE; ++i) {
            max_val = max(max_val, vals[i]);
        }
    }
    // Handle remaining elements
    for (int idx = vec_dim * VEC_SIZE + tid; idx < dim_size; idx += blockDim.x) {
        max_val = max(max_val, __ldg(input_row + idx));
    }

    max_val = warp_reduce_max(max_val);
    if (lane_id == 0) warp_max[warp_id] = max_val;
    __syncthreads();

    if (tid < 32) {
        max_val = (tid < warps_per_block) ? warp_max[tid] : -std::numeric_limits<scalar_t>::infinity();
        max_val = warp_reduce_max(max_val);
        if (tid == 0) warp_max[0] = max_val;
    }
    __syncthreads();
    max_val = warp_max[0];

    // Vectorized sum reduction
    scalar_t sum = 0;
    for (int idx = tid; idx < vec_dim; idx += blockDim.x) {
        scalar_t vals[VEC_SIZE];
        *reinterpret_cast<typename std::conditional<VEC_SIZE == 4, float4, double2>::type*>(vals) = 
            *reinterpret_cast<const typename std::conditional<VEC_SIZE == 4, float4, double2>::type*>(input_row + idx * VEC_SIZE);
        
        #pragma unroll
        for (int i = 0; i < VEC_SIZE; ++i) {
            sum += exp(vals[i] - max_val);
        }
    }
    // Handle remaining elements
    for (int idx = vec_dim * VEC_SIZE + tid; idx < dim_size; idx += blockDim.x) {
        sum += exp(__ldg(input_row + idx) - max_val);
    }

    sum = warp_reduce_sum(sum);
    if (lane_id == 0) warp_sum[warp_id] = sum;
    __syncthreads();

    if (tid < 32) {
        sum = (tid < warps_per_block) ? warp_sum[tid] : 0;
        sum = warp_reduce_sum(sum);
        if (tid == 0) warp_sum[0] = sum;
    }
    __syncthreads();
    scalar_t log_sum = log(warp_sum[0]);

    // Vectorized write output
    for (int idx = tid; idx < vec_dim; idx += blockDim.x) {
        scalar_t out_vals[VEC_SIZE];
        const scalar_t* in_ptr = input_row + idx * VEC_SIZE;
        
        #pragma unroll
        for (int i = 0; i < VEC_SIZE; ++i) {
            out_vals[i] = (__ldg(in_ptr + i) - max_val) - log_sum;
        }
        
        *reinterpret_cast<typename std::conditional<VEC_SIZE == 4, float4, double2>::type*>(output_row + idx * VEC_SIZE) = 
            *reinterpret_cast<typename std::conditional<VEC_SIZE == 4, float4, double2>::type*>(out_vals);
    }
    // Handle remaining elements
    for (int idx = vec_dim * VEC_SIZE + tid; idx < dim_size; idx += blockDim.x) {
        output_row[idx] = (__ldg(input_row + idx) - max_val) - log_sum;
    }
}

torch::Tensor log_softmax_cuda_forward(torch::Tensor input, int64_t dim) {
    auto ndim = input.dim();
    dim = dim >= 0 ? dim : dim + ndim;

    std::vector<int64_t> permute_dims;
    for (int64_t i = 0; i < ndim; ++i) {
        if (i != dim) permute_dims.push_back(i);
    }
    permute_dims.push_back(dim);

    input = input.permute(permute_dims).contiguous();
    auto output = torch::empty_like(input);
    
    int64_t batch_size = input.numel() / input.size(-1);
    int64_t dim_size = input.size(-1);

    const int threads = std::min(1024, static_cast<int>(((dim_size + 31)/32)*32));
    const bool use_float4 = input.scalar_type() == torch::kFloat32 && dim_size % 4 == 0;
    const bool use_double2 = input.scalar_type() == torch::kFloat64 && dim_size % 2 == 0;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "log_softmax_forward_cuda", ([&] {
        size_t shared_mem_size = (threads/32) * sizeof(scalar_t) * 2;
        if (use_float4) {
            log_softmax_forward_kernel<scalar_t, 4><<<batch_size, threads, shared_mem_size>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else if (use_double2) {
            log_softmax_forward_kernel<scalar_t, 2><<<batch_size, threads, shared_mem_size>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else {
            log_softmax_forward_kernel<scalar_t, 1><<<batch_size, threads, shared_mem_size>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        }
    }));

    std::vector<int64_t> inverse_permute_dims(ndim);
    for (size_t i = 0; i < permute_dims.size(); ++i) {
        inverse_permute_dims[permute_dims[i]] = i;
    }
    return output.permute(inverse_permute_dims);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &log_softmax_cuda_forward, "LogSoftmax forward (CUDA)");
}
