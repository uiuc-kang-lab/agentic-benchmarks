#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void conv_transpose1d_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ y,
    int N, int C_in, int C_out, int L_in, int L_out, int K_w,
    int stride, int padding, int dilation) {
    
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int num_threads = blockDim.x * gridDim.x;
    const int total_elements = N * C_out * L_out;

    // Stride loop to handle multiple elements per thread
    for (int idx = tid; idx < total_elements; idx += num_threads) {
        const int l_out = idx % L_out;
        const int c_out = (idx / L_out) % C_out;
        const int n = idx / (L_out * C_out);

        float value = bias != nullptr ? bias[c_out] : 0.0f;

        #pragma unroll 4
        for (int c_in = 0; c_in < C_in; ++c_in) {
            const int x_batch_offset = n * C_in * L_in + c_in * L_in;
            const int w_channel_offset = c_in * C_out * K_w + c_out * K_w;

            for (int k_w = 0; k_w < K_w; ++k_w) {
                const int l_in_nom = l_out + padding - k_w * dilation;
                if (l_in_nom % stride == 0) {
                    const int l_in = l_in_nom / stride;
                    if (l_in >= 0 && l_in < L_in) {
                        value += x[x_batch_offset + l_in] * 
                                weight[w_channel_offset + k_w];
                    }
                }
            }
        }
        y[n * C_out * L_out + c_out * L_out + l_out] = value;
    }
}

torch::Tensor conv_transpose1d_forward(
    py::object x_obj,
    py::object weight_obj,
    py::object bias_obj = py::none(),
    int64_t stride = 1,
    int64_t padding = 0,
    int64_t dilation = 1) {

    torch::Tensor x = x_obj.cast<torch::Tensor>();
    torch::Tensor weight = weight_obj.cast<torch::Tensor>();

    x = x.contiguous();
    weight = weight.contiguous();

    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA device");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA device");

    float* bias_ptr = nullptr;
    if (!bias_obj.is_none()) {
        torch::Tensor bias = bias_obj.cast<torch::Tensor>();
        bias = bias.contiguous();
        TORCH_CHECK(bias.is_cuda(), "Bias tensor must be on CUDA device");
        bias_ptr = bias.data_ptr<float>();
    }

    int N = x.size(0);
    int C_in = x.size(1);
    int L_in = x.size(2);
    int K_w = weight.size(2);
    int C_out = weight.size(1);
    int L_out = (L_in - 1) * stride - 2 * padding + dilation * (K_w - 1) + 1;

    auto y = torch::empty({N, C_out, L_out}, x.options());

    const int threads = 256;
    const int max_blocks = 65535;
    const int min_elements_per_thread = 4;
    const int total_elements = N * C_out * L_out;
    
    int blocks = std::min(max_blocks, 
                         (total_elements + threads * min_elements_per_thread - 1) / 
                         (threads * min_elements_per_thread));

    conv_transpose1d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        y.data_ptr<float>(),
        N, C_in, C_out, L_in, L_out, K_w,
        stride, padding, dilation);

    TORCH_CHECK(hipGetLastError() == hipSuccess, "CUDA kernel failed");

    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        &conv_transpose1d_forward,
        "Conv Transpose1D forward (CUDA)",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = 1,
        py::arg("padding") = 0,
        py::arg("dilation") = 1);
}