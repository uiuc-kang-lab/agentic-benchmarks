#include "hip/hip_runtime.h"
/* 
   Unified CUDA extension for ConvTranspose2d
   Combines PyTorch extension boilerplate with an optimized CUDA kernel that
   manually unrolls loops for the common 3x3 kernel case, and falls back to a
   generic loop implementation for other kernel sizes.
*/

#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

namespace py = pybind11;

// Utility: Parse int or sequence of ints from Python objects
inline std::vector<int64_t> parseIntArrayRef(const py::object &obj) {
    std::vector<int64_t> result;
    if (py::isinstance<py::int_>(obj)) {
        result.push_back(obj.cast<int64_t>());
    } else if (py::isinstance<py::sequence>(obj)) {
        for (auto item : obj.cast<py::sequence>()) {
            result.push_back(py::cast<int64_t>(item));
        }
    } else {
        throw std::runtime_error("Expected int or sequence of ints");
    }
    return result;
}

// Unified CUDA kernel for ConvTranspose2d with manual unrolling when kernel_size==3
__global__ void conv_transpose2d_unified_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,  // may be nullptr
    float* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int h_in,
    const int w_in,
    const int out_channels,
    const int h_out,
    const int w_out,
    const int kernel_size,
    const int stride_h,
    const int stride_w,
    const int padding_h,
    const int padding_w,
    const int groups,
    const int in_channels_per_group,
    const int out_channels_per_group
) {
    int total = batch_size * out_channels * h_out * w_out;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < total;
         index += blockDim.x * gridDim.x) {
        // Compute output indices: n, c, h, w
        int w = index % w_out;
        int tmp = index / w_out;
        int h = tmp % h_out;
        tmp = tmp / h_out;
        int c = tmp % out_channels;
        int n = tmp / out_channels;

        int g = c / out_channels_per_group;  // group index
        int c_local = c % out_channels_per_group;
        float sum = 0.0f;

        if (kernel_size == 3) {
            // Optimized manual unrolling for 3x3 kernels
            // Loop over kernel height (kh) unrolled
            // Note: We compute candidate input positions and check divisibility by stride.
            // Unroll kh = 0, 1, 2
            for (int kh = 0; kh < 3; ++kh) {
                int h_in_candidate = h + padding_h - kh;
                if ((h_in_candidate % stride_h) == 0) {
                    int h_in_idx = h_in_candidate / stride_h;
                    if (h_in_idx >= 0 && h_in_idx < h_in) {
                        // Manually unroll kw = 0,1,2
                        {   // kw = 0
                            int kw = 0;
                            int w_in_candidate = w + padding_w - kw;
                            if ((w_in_candidate % stride_w) == 0) {
                                int w_in_idx = w_in_candidate / stride_w;
                                if (w_in_idx >= 0 && w_in_idx < w_in) {
                                    for (int r = 0; r < in_channels_per_group; ++r) {
                                        int in_channel = g * in_channels_per_group + r;
                                        int input_idx = ((n * in_channels + in_channel) * h_in + h_in_idx) * w_in + w_in_idx;
                                        int weight_idx = (((g * in_channels_per_group + r) * out_channels_per_group + c_local) * kernel_size + kh) * kernel_size + kw;
                                        sum += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
                                    }
                                }
                            }
                        }
                        {   // kw = 1
                            int kw = 1;
                            int w_in_candidate = w + padding_w - kw;
                            if ((w_in_candidate % stride_w) == 0) {
                                int w_in_idx = w_in_candidate / stride_w;
                                if (w_in_idx >= 0 && w_in_idx < w_in) {
                                    for (int r = 0; r < in_channels_per_group; ++r) {
                                        int in_channel = g * in_channels_per_group + r;
                                        int input_idx = ((n * in_channels + in_channel) * h_in + h_in_idx) * w_in + w_in_idx;
                                        int weight_idx = (((g * in_channels_per_group + r) * out_channels_per_group + c_local) * kernel_size + kh) * kernel_size + kw;
                                        sum += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
                                    }
                                }
                            }
                        }
                        {   // kw = 2
                            int kw = 2;
                            int w_in_candidate = w + padding_w - kw;
                            if ((w_in_candidate % stride_w) == 0) {
                                int w_in_idx = w_in_candidate / stride_w;
                                if (w_in_idx >= 0 && w_in_idx < w_in) {
                                    for (int r = 0; r < in_channels_per_group; ++r) {
                                        int in_channel = g * in_channels_per_group + r;
                                        int input_idx = ((n * in_channels + in_channel) * h_in + h_in_idx) * w_in + w_in_idx;
                                        int weight_idx = (((g * in_channels_per_group + r) * out_channels_per_group + c_local) * kernel_size + kh) * kernel_size + kw;
                                        sum += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
                                    }
                                }
                            }
                        }
                    }
                }
            }
        } else {
            // Generic loop for arbitrary kernel_size
            for (int kh = 0; kh < kernel_size; ++kh) {
                int h_in_candidate = h + padding_h - kh;
                if ((h_in_candidate % stride_h) != 0) continue;
                int h_in_idx = h_in_candidate / stride_h;
                if (h_in_idx < 0 || h_in_idx >= h_in) continue;
                for (int kw = 0; kw < kernel_size; ++kw) {
                    int w_in_candidate = w + padding_w - kw;
                    if ((w_in_candidate % stride_w) != 0) continue;
                    int w_in_idx = w_in_candidate / stride_w;
                    if (w_in_idx < 0 || w_in_idx >= w_in) continue;
                    for (int r = 0; r < in_channels_per_group; ++r) {
                        int in_channel = g * in_channels_per_group + r;
                        int input_idx = ((n * in_channels + in_channel) * h_in + h_in_idx) * w_in + w_in_idx;
                        int weight_idx = (((g * in_channels_per_group + r) * out_channels_per_group + c_local) * kernel_size + kh) * kernel_size + kw;
                        sum += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
                    }
                }
            }
        }

        // Add bias if provided
        if (bias != nullptr) {
            sum += __ldg(&bias[c]);
        }
        int output_idx = ((n * out_channels + c) * h_out + h) * w_out + w;
        output[output_idx] = sum;
    }
}

// Forward function: prepares tensor dimensions and launches the unified kernel
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    py::object stride = py::int_(1),
    py::object padding = py::int_(0),
    py::object output_padding = py::int_(0),
    int64_t groups = 1
) {
    auto stride_vec = parseIntArrayRef(stride);
    auto padding_vec = parseIntArrayRef(padding);
    auto output_padding_vec = parseIntArrayRef(output_padding);

    int stride_h = stride_vec[0];
    int stride_w = (stride_vec.size() > 1) ? stride_vec[1] : stride_h;
    int padding_h = padding_vec[0];
    int padding_w = (padding_vec.size() > 1) ? padding_vec[1] : padding_h;
    int output_padding_h = output_padding_vec[0];
    int output_padding_w = (output_padding_vec.size() > 1) ? output_padding_vec[1] : output_padding_h;

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int h_in = x.size(2);
    const int w_in = x.size(3);

    const int kernel_size = weight.size(2);
    int out_channels = weight.size(1) * groups;

    // Calculate output dimensions
    int h_out = (h_in - 1) * stride_h - 2 * padding_h + kernel_size + output_padding_h;
    int w_out = (w_in - 1) * stride_w - 2 * padding_w + kernel_size + output_padding_w;

    auto output_tensor = torch::zeros({batch_size, out_channels, h_out, w_out}, x.options());

    int in_channels_per_group = in_channels / groups;
    int out_channels_per_group = out_channels / groups;

    int total_elements = batch_size * out_channels * h_out * w_out;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;

    const float* input_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float* output_ptr = output_tensor.data_ptr<float>();

    conv_transpose2d_unified_kernel<<<grid_size, block_size>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch_size,
        in_channels,
        h_in,
        w_in,
        out_channels,
        h_out,
        w_out,
        kernel_size,
        stride_h,
        stride_w,
        padding_h,
        padding_w,
        groups,
        in_channels_per_group,
        out_channels_per_group
    );

    hipDeviceSynchronize();
    return output_tensor;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Unified ConvTranspose2d forward kernel (optimized for 3x3 kernels)",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride") = 1,
          py::arg("padding") = 0,
          py::arg("output_padding") = 0,
          py::arg("groups") = 1);
}
