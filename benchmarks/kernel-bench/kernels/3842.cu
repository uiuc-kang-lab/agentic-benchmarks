#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_softplus(scalar_t x) {
    if (x > 20.0) {
        return x;
    } else if (x < -20.0) {
        return __expf(x);
    } else {
        const scalar_t exp_x = __expf(x);
        return log1p(exp_x);
    }
}

template <typename scalar_t>
__global__ void softplus_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int elements_per_thread = 8;
    
    #pragma unroll
    for (int i = 0; i < elements_per_thread; i++) {
        const int idx = tid + i * stride;
        if (idx < size) {
            const scalar_t x = input[idx];
            output[idx] = compute_softplus(x);
        }
    }
}

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    const int threads = 256;
    // Adjust block count to account for each thread handling 8 elements
    const int blocks = min(65535, (size + (threads * 8) - 1) / (threads * 8));

    AT_DISPATCH_FLOATING_TYPES(input.type(), "softplus_forward_cuda", ([&] {
        softplus_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}