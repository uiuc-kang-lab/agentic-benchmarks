#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Tile size used for shared memory tiling
#define TILE_SIZE 16

// Optimized kernel using shared memory tiling with __ldg() for read-only accesses
__global__ void matrix_mul_ldg_kernel(const float* __restrict__ A,
                                        const float* __restrict__ B,
                                        float* __restrict__ C,
                                        int M, int N, int K) {
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0f;

    // Loop over tiles along the K dimension
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; t++) {
        int tiledX = t * TILE_SIZE + threadIdx.x;
        int tiledY = t * TILE_SIZE + threadIdx.y;

        // Load element of A using __ldg() if within bounds; assumes A is 128-bit aligned
        if (row < M && tiledX < K)
            tile_A[threadIdx.y][threadIdx.x] = __ldg(&A[row * K + tiledX]);
        else
            tile_A[threadIdx.y][threadIdx.x] = 0.0f;

        // Load element of B using __ldg() if within bounds; assumes B is 128-bit aligned
        if (tiledY < K && col < N)
            tile_B[threadIdx.y][threadIdx.x] = __ldg(&B[tiledY * N + col]);
        else
            tile_B[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Compute partial multiplication for the current tile
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += tile_A[threadIdx.y][k] * tile_B[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

// Launcher function to execute the kernel
void matrix_multiply_cuda(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const float* d_A = A.data_ptr<float>();
    const float* d_B = B.data_ptr<float>();
    float* d_C = C.data_ptr<float>();

    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);

    matrix_mul_ldg_kernel<<<grid, block>>>(d_A, d_B, d_C, M, N, K);
    hipDeviceSynchronize();
}

// Pybind interface
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());
    matrix_multiply_cuda(A, B, C);
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication with read-only __ldg() and aligned loads (CUDA)");
}
