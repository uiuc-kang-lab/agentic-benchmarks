#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#define WARP_SIZE 32
#define BLOCK_SIZE 256
#define ELEMENTS_PER_THREAD 4
#define CHUNK_SIZE (1 << 16)  // 65536 elements per chunk
#define STREAM_COUNT 4
#define MIN_ELEMENTS_FOR_STREAMING (1 << 22)  // 4M elements threshold

__device__ __forceinline__ float warp_reduce(float val) {
    #pragma unroll
    for (int offset = WARP_SIZE/2; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__global__ void kl_div_kernel_optimized(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets,
    float* __restrict__ output,
    const int n,
    const int elements_per_thread) {
    
    const int tid = threadIdx.x;
    const int wid = tid / WARP_SIZE;
    const int lane = tid % WARP_SIZE;
    const int global_thread_id = blockIdx.x * blockDim.x + tid;
    
    extern __shared__ float warp_results[];
    
    float thread_sum = 0.0f;
    
    // Each thread processes multiple elements with coalesced memory access
    const int start_idx = global_thread_id * elements_per_thread;
    #pragma unroll
    for (int i = 0; i < elements_per_thread; i++) {
        const int idx = start_idx + i;
        if (idx < n) {
            const float log_pred = log_predictions[idx];
            const float target = targets[idx];
            thread_sum += __expf(log_pred) - target * log_pred;  // Using fast math
        }
    }
    
    // Two-level reduction: first within warps, then across warps
    thread_sum = warp_reduce(thread_sum);
    
    if (lane == 0) {
        warp_results[wid] = thread_sum;
    }
    __syncthreads();
    
    if (wid == 0) {
        float warp_sum = (lane < (BLOCK_SIZE / WARP_SIZE)) ? warp_results[lane] : 0.0f;
        warp_sum = warp_reduce(warp_sum);
        
        if (lane == 0) {
            atomicAdd(output, warp_sum);
        }
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options().device(torch::kCUDA));

    if (!log_predictions.is_cuda() && n >= MIN_ELEMENTS_FOR_STREAMING) {
        hipStream_t streams[STREAM_COUNT];
        for (int i = 0; i < STREAM_COUNT; i++) {
            hipStreamCreate(&streams[i]);
        }

        float* h_log_predictions = log_predictions.data_ptr<float>();
        float* h_targets = targets.data_ptr<float>();
        
        int offset = 0;
        while (offset < n) {
            int current_chunk = std::min(CHUNK_SIZE, n - offset);
            int stream_idx = (offset / CHUNK_SIZE) % STREAM_COUNT;
            hipStream_t stream = streams[stream_idx];

            float* d_log_chunk = nullptr;
            float* d_target_chunk = nullptr;
            hipMallocAsync((void**)&d_log_chunk, current_chunk * sizeof(float), stream);
            hipMallocAsync((void**)&d_target_chunk, current_chunk * sizeof(float), stream);

            hipMemcpyAsync(d_log_chunk, h_log_predictions + offset,
                          current_chunk * sizeof(float), hipMemcpyHostToDevice, stream);
            hipMemcpyAsync(d_target_chunk, h_targets + offset,
                          current_chunk * sizeof(float), hipMemcpyHostToDevice, stream);

            const int elements_per_thread = ELEMENTS_PER_THREAD;
            const int total_threads_needed = (current_chunk + elements_per_thread - 1) / elements_per_thread;
            const int blocks = (total_threads_needed + BLOCK_SIZE - 1) / BLOCK_SIZE;
            const int warps_per_block = BLOCK_SIZE / WARP_SIZE;
            const int shared_mem = warps_per_block * sizeof(float);

            kl_div_kernel_optimized<<<blocks, BLOCK_SIZE, shared_mem, stream>>>(
                d_log_chunk, d_target_chunk, output.data_ptr<float>(),
                current_chunk, elements_per_thread);

            hipFreeAsync(d_log_chunk, stream);
            hipFreeAsync(d_target_chunk, stream);
            offset += current_chunk;
        }

        for (int i = 0; i < STREAM_COUNT; i++) {
            hipStreamSynchronize(streams[i]);
            hipStreamDestroy(streams[i]);
        }
    } else {
        const int elements_per_thread = ELEMENTS_PER_THREAD;
        const int total_threads_needed = (n + elements_per_thread - 1) / elements_per_thread;
        const int blocks = (total_threads_needed + BLOCK_SIZE - 1) / BLOCK_SIZE;
        const int warps_per_block = BLOCK_SIZE / WARP_SIZE;
        const int shared_mem = warps_per_block * sizeof(float);

        kl_div_kernel_optimized<<<blocks, BLOCK_SIZE, shared_mem>>>(
            log_predictions.data_ptr<float>(),
            targets.data_ptr<float>(),
            output.data_ptr<float>(),
            n,
            elements_per_thread
        );
    }
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "Adaptive KL divergence forward (CUDA)");
}