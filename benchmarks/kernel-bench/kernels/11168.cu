#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void smooth_l1_loss_optimized_kernel(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    int n_elements
) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;
    int stride = gridDim.x * blockDim.x;
    float thread_sum = 0.0f;

    // Vectorized processing
    int vec_count = n_elements / 4;
    const float4* pred4 = reinterpret_cast<const float4*>(predictions);
    const float4* targ4 = reinterpret_cast<const float4*>(targets);

    for (int i = idx; i < vec_count; i += stride) {
        float4 p = __ldg(pred4 + i);
        float4 t = __ldg(targ4 + i);

        float diff = p.x - t.x;
        thread_sum += (fabsf(diff) < 1.0f) ? 0.5f*diff*diff : fabsf(diff)-0.5f;
        
        diff = p.y - t.y;
        thread_sum += (fabsf(diff) < 1.0f) ? 0.5f*diff*diff : fabsf(diff)-0.5f;
        
        diff = p.z - t.z;
        thread_sum += (fabsf(diff) < 1.0f) ? 0.5f*diff*diff : fabsf(diff)-0.5f;
        
        diff = p.w - t.w;
        thread_sum += (fabsf(diff) < 1.0f) ? 0.5f*diff*diff : fabsf(diff)-0.5f;
    }

    // Scalar processing for remainder
    int scalar_base = vec_count * 4;
    for (int i = scalar_base + idx; i < n_elements; i += stride) {
        float diff = __ldg(predictions + i) - __ldg(targets + i);
        thread_sum += (fabsf(diff) < 1.0f) ? 0.5f*diff*diff : fabsf(diff)-0.5f;
    }

    // Optimized block reduction
    __shared__ float shared_mem[256];
    shared_mem[tid] = thread_sum;
    __syncthreads();

    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_mem[tid] += shared_mem[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(output, shared_mem[0] / n_elements);
    }
}

torch::Tensor smooth_l1_loss_optimized(
    torch::Tensor predictions,
    torch::Tensor targets
) {
    TORCH_CHECK(predictions.sizes() == targets.sizes(), "Input shape mismatch");
    TORCH_CHECK(predictions.is_contiguous() && targets.is_contiguous(), "Non-contiguous inputs");

    int n = predictions.numel();
    auto output = torch::zeros({1}, predictions.options());

    const int block_size = 256;
    int grid_size = (n / 4 + block_size - 1) / block_size;
    grid_size = grid_size > 0 ? grid_size : 1;

    smooth_l1_loss_optimized_kernel<<<grid_size, block_size>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &smooth_l1_loss_optimized, "Optimized Smooth L1 Loss");
}