#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Device function for performing the convolution
__device__ void conv_transpose3d_computation(
    const float* __restrict__ input, const float* __restrict__ weight,
    float& sum, int n, int ic, int oc, int D, int H, int W,
    int id, int ih, int iw, int kd, int kh, int kw,
    int stride_d, int stride_h, int stride_w,
    int iD, int iH, int iW,
    int kD, int kH, int kW) {
    // Iterate through kernel dimensions
    #pragma unroll 8
    for (int kd_unrolled = 0; kd_unrolled < kD; ++kd_unrolled) {
        int id_candidate = id + kd - kd_unrolled * stride_d;
        if (id_candidate < 0 || id_candidate >= iD) continue;
        #pragma unroll 8
        for (int kh_unrolled = 0; kh_unrolled < kH; ++kh_unrolled) {
            int ih_candidate = ih + kh - kh_unrolled * stride_h;
            if (ih_candidate < 0 || ih_candidate >= iH) continue;
            #pragma unroll 8
            for (int kw_unrolled = 0; kw_unrolled < kW; ++kw_unrolled) {
                int iw_candidate = iw + kw - kw_unrolled * stride_w;
                if (iw_candidate < 0 || iw_candidate >= iW) continue;

                // Calculate input and weight index
                int input_idx = (((n * iD + id_candidate) * iH + ih_candidate) * iW + iw_candidate);
                int weight_idx = (((((oc * iD) + kd_unrolled) * kH + kh_unrolled) * kW + kw_unrolled) * iD + ic);
                
                // Accumulate sum
                sum += input[input_idx] * weight[weight_idx];
            }
        }
    }
}

// CUDA kernel rewritten to use a modular computation function
__global__ void conv_transpose3d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,  // can be nullptr
    float* __restrict__ output,
    int batch,
    int in_channels,
    int out_channels,
    int iD, int iH, int iW,
    int kD, int kH, int kW,
    int stride_d, int stride_h, int stride_w,
    int pad_d, int pad_h, int pad_w,
    int outD, int outH, int outW,
    int groups) {

    int total_elements = batch * out_channels * outD * outH * outW;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= total_elements) return;

    // Decode output index into (b, oc, d, h, w)
    int w = index % outW;
    int tmp = index / outW;
    int h = tmp % outH;
    tmp = tmp / outH;
    int d = tmp % outD;
    tmp = tmp / outD;
    int oc = tmp % out_channels;
    int b = tmp / out_channels;

    float sum = 0.0f;

    // Case for non-grouped convolution
    if (groups == 1) {
        for (int ic = 0; ic < in_channels; ++ic) {
            conv_transpose3d_computation(
                input, weight, sum, b, ic, oc, d, h, w, d, h, w, kD, kH, kW,
                stride_d, stride_h, stride_w, iD, iH, iW, kD, kH, kW);
        }
    } else {
        // Grouped convolution
        int out_channels_per_group = out_channels / groups;
        int in_channels_per_group = in_channels / groups;
        int group = oc / out_channels_per_group;
        for (int ic = group * in_channels_per_group; ic < (group + 1) * in_channels_per_group; ++ic) {
            conv_transpose3d_computation(
                input, weight, sum, b, ic, oc, d, h, w, d, h, w, kD, kH, kW,
                stride_d, stride_h, stride_w, iD, iH, iW, kD, kH, kW);
        }
    }

    // Add bias if provided
    if (bias != nullptr) {
        sum += bias[oc];
    }
    // Write the result into output tensor
    output[(((b * out_channels + oc) * outD + d) * outH + h) * outW + w] = sum;
}

// Host forward function that prepares parameters and launches the kernel
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> output_padding,
    int64_t groups
) {
    // Input dimensions
    int batch = x.size(0);
    int in_channels = x.size(1);
    int iD = x.size(2);
    int iH = x.size(3);
    int iW = x.size(4);

    // Weight dimensions
    int kD = weight.size(2);
    int kH = weight.size(3);
    int kW = weight.size(4);

    int stride_d = stride[0];
    int stride_h = stride[1];
    int stride_w = stride[2];

    int pad_d = padding[0];
    int pad_h = padding[1];
    int pad_w = padding[2];

    int out_pad_d = output_padding[0];
    int out_pad_h = output_padding[1];
    int out_pad_w = output_padding[2];

    // Compute output dimensions
    int outD = (iD - 1) * stride_d - 2 * pad_d + kD + out_pad_d;
    int outH = (iH - 1) * stride_h - 2 * pad_h + kH + out_pad_h;
    int outW = (iW - 1) * stride_w - 2 * pad_w + kW + out_pad_w;

    int out_channels = (groups == 1) ? weight.size(1) : weight.size(1) * groups;

    auto options = x.options();
    auto output = torch::zeros({batch, out_channels, outD, outH, outW}, options);

    int total_threads = batch * out_channels * outD * outH * outW;
    int threads = 256;
    int blocks = (total_threads + threads - 1) / threads;

    const float* x_ptr = x.data_ptr<float>();
    const float* w_ptr = weight.data_ptr<float>();
    const float* b_ptr = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float* out_ptr = output.data_ptr<float>();

    conv_transpose3d_kernel<<<blocks, threads>>>(x_ptr, w_ptr, b_ptr, out_ptr,
        batch, in_channels, out_channels,
        iD, iH, iW,
        kD, kH, kW,
        stride_d, stride_h, stride_w,
        pad_d, pad_h, pad_w,
        outD, outH, outW,
        groups);

    hipDeviceSynchronize();
    return output;
}

// PyBind11 module definition
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ConvTranspose3d forward function with modular device functions",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = nullptr,
          py::arg("stride"),
          py::arg("padding"),
          py::arg("output_padding"),
          py::arg("groups"));
}