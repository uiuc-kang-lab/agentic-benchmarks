#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32
#define MAX_DIAG_SIZE 16384  // 64KB / sizeof(float)

// Constant memory for diagonal matrix
__constant__ float d_diag[MAX_DIAG_SIZE];

// CUDA kernel using constant memory and vectorized loads
__global__ void diag_matmul_kernel_constant_mem(
    const float* __restrict__ B,
    float* __restrict__ C,
    const int64_t N,
    const int64_t M
) {
    int lane = threadIdx.x % WARP_SIZE;
    int warpId = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;

    if (warpId < N) {
        const float a_val = d_diag[warpId];  // Load from constant memory
        const int row_offset = warpId * M;

        // Use vectorized memory operations if M is divisible by 4
        if (M % 4 == 0) {
            int vec_M = M / 4;
            for (int vec = lane; vec < vec_M; vec += WARP_SIZE) {
                int idx = row_offset / 4 + vec;
                float4 b_val = reinterpret_cast<const float4*>(B)[idx];
                float4 c_val;
                c_val.x = a_val * b_val.x;
                c_val.y = a_val * b_val.y;
                c_val.z = a_val * b_val.z;
                c_val.w = a_val * b_val.w;
                reinterpret_cast<float4*>(C)[idx] = c_val;
            }
        } else {
            // Fallback to scalar operations
            for (int col = lane; col < M; col += WARP_SIZE) {
                int idx = row_offset + col;
                C[idx] = a_val * B[idx];
            }
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.dim() == 1, "A must be a 1D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == B.size(0), "Dimension mismatch: A.size(0) must match B.size(0)");
    TORCH_CHECK(A.size(0) <= MAX_DIAG_SIZE, "Diagonal matrix too large for constant memory");

    A = A.contiguous();
    B = B.contiguous();

    int64_t N = A.size(0);
    int64_t M = B.size(1);

    auto C = torch::empty({N, M}, B.options());

    // Copy diagonal matrix to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_diag), A.data_ptr<float>(), N * sizeof(float));

    // Configure kernel
    int threadsPerBlock = 128;
    int warpsPerBlock = threadsPerBlock / WARP_SIZE;
    int blocks = (N + warpsPerBlock - 1) / warpsPerBlock;

    diag_matmul_kernel_constant_mem<<<blocks, threadsPerBlock>>>(
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N,
        M
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Diagonal matrix multiplication using constant memory");
}