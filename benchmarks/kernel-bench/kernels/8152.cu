#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel using shared memory to optimize memory access patterns

template <typename scalar_t>
__global__ void conv_transpose2d_kernel_shared_memory(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int out_channels,
    const int kernel_h,
    const int kernel_w,
    const int stride,
    const int padding,
    const int output_padding,
    const int groups,
    const int dilation,
    const int out_height,
    const int out_width
) {
    extern __shared__ scalar_t shared_input[];  // Shared memory for input tiles
    const int total_elements = batch_size * out_channels * out_height * out_width;
    const int gridStride = blockDim.x * gridDim.x;

    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < total_elements; idx += gridStride) {
        int n = idx;
        const int ow = n % out_width;
        n /= out_width;
        const int oh = n % out_height;
        n /= out_height;
        const int oc = n % out_channels;
        n /= out_channels;
        const int b = n;

        if (b >= batch_size) continue;

        const int out_channels_per_group = out_channels / groups;
        const int g = oc / out_channels_per_group;
        const int oc_group = oc % out_channels_per_group;
        const int in_channels_per_group = in_channels / groups;
        const int ic_start = g * in_channels_per_group;

        scalar_t val = (bias != nullptr) ? bias[oc] : static_cast<scalar_t>(0);

        for (int kh = 0; kh < kernel_h; ++kh) {
            int h_in_temp = oh - kh * dilation + padding;
            if (h_in_temp % stride != 0) continue;
            int h_in = h_in_temp / stride;
            if (h_in < 0 || h_in >= in_height) continue;

            for (int kw = 0; kw < kernel_w; ++kw) {
                int w_in_temp = ow - kw * dilation + padding;
                if (w_in_temp % stride != 0) continue;
                int w_in = w_in_temp / stride;
                if (w_in < 0 || w_in >= in_width) continue;

                // Load input data into shared memory
                int shared_idx = threadIdx.x;
                for (int ic = 0; ic < in_channels_per_group; ++ic) {
                    if (shared_idx < in_channels_per_group * in_height * in_width) {
                        shared_input[shared_idx] = input[b * (in_channels * in_height * in_width)
                                                         + (ic_start + ic) * (in_height * in_width)
                                                         + h_in * in_width + w_in];
                    }
                    shared_idx += blockDim.x;
                }
                __syncthreads();  // Ensure all data is loaded before computation

                // Perform convolution using shared memory
                for (int ic = 0; ic < in_channels_per_group; ++ic) {
                    int weight_idx = (ic_start + ic) * (out_channels_per_group * kernel_h * kernel_w)
                                     + oc_group * (kernel_h * kernel_w)
                                     + kh * kernel_w + kw;
                    val += shared_input[ic * (in_height * in_width) + h_in * in_width + w_in] * weight[weight_idx];
                }
                __syncthreads();  // Ensure completion of computation before next iteration
            }
        }

        output[idx] = val;
    }
}

// Forward function that sets up kernel parameters

torch::Tensor forward(
    torch::Tensor input,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int output_padding,
    int groups,
    int dilation = 1
) {
    TORCH_CHECK(input.device().is_cuda(), "input must be a CUDA tensor");
    TORCH_CHECK(input.dim() == 4, "Input must be 4D");
    TORCH_CHECK(weight.dim() == 4, "Weight must be 4D");

    const int batch_size = input.size(0);
    const int in_channels = input.size(1);
    const int in_height = input.size(2);
    const int in_width = input.size(3);

    const int out_channels = weight.size(1) * groups;
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);

    if (bias.has_value() && bias->defined()) {
        TORCH_CHECK(bias->numel() == out_channels, "Bias must have out_channels elements");
        TORCH_CHECK(bias->device().is_cuda(), "Bias must be a CUDA tensor");
    }

    const int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_h - 1) + output_padding + 1;
    const int out_width = (in_width - 1) * stride - 2 * padding + dilation * (kernel_w - 1) + output_padding + 1;

    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());

    const int total_elements = output.numel();
    constexpr int THREADS = 256;
    const int BLOCKS = (total_elements + THREADS - 1) / THREADS;
    const int shared_mem_size = in_channels * in_height * in_width * sizeof(scalar_t);

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "conv_transpose2d_cuda_shared_memory", ([&] {
        conv_transpose2d_kernel_shared_memory<scalar_t><<<BLOCKS, THREADS, shared_mem_size>>>(
            input.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            (bias.has_value() && bias->defined()) ? bias->data_ptr<scalar_t>() : nullptr,
            output.data_ptr<scalar_t>(),
            batch_size,
            in_channels,
            in_height,
            in_width,
            out_channels,
            kernel_h,
            kernel_w,
            stride,
            padding,
            output_padding,
            groups,
            dilation,
            out_height,
            out_width
        );
    }));

    return output;
}

// Pybind the forward function

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Shared Memory Transposed 2D Convolution (CUDA)",
          py::arg("input"), py::arg("weight"), py::arg("bias") = py::none(),
          py::arg("stride"), py::arg("padding"), py::arg("output_padding"),
          py::arg("groups"), py::arg("dilation") = 1);
}
