#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdio>

// Define block size macro for tuning. This can be overridden via compile flags (e.g., -DBLOCK_SIZE=128)
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256
#endif

// Optimized CUDA kernel for 2D convolution using tuned block size and memory coalescing
__global__ void conv2d_cuda_kernel_coalesced(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int N, int C_in, int H_in, int W_in,
    int C_out, int H_out, int W_out,
    int K_h, int K_w,
    int stride_h, int stride_w,
    int padding_h, int padding_w,
    int dilation_h, int dilation_w,
    int groups
) {
    int n = blockIdx.x;
    int c_out = blockIdx.y;
    int h_out = blockIdx.z * blockDim.y + threadIdx.y;
    int w_out = threadIdx.x;

    float value = (bias != nullptr) ? bias[c_out] : 0.0f;

    int group = c_out / (C_out / groups);
    int c_in_start = group * (C_in / groups);
    int c_in_end = c_in_start + (C_in / groups);

    for (int c_in = c_in_start; c_in < c_in_end; ++c_in) {
        for (int k_h = 0; k_h < K_h; ++k_h) {
            int h_in = h_out * stride_h - padding_h + k_h * dilation_h;
            if (h_in >= 0 && h_in < H_in) {
                for (int k_w = 0; k_w < K_w; ++k_w) {
                    int w_in = w_out * stride_w - padding_w + k_w * dilation_w;
                    if (w_in >= 0 && w_in < W_in) {
                        int input_idx = ((n * C_in + c_in) * H_in + h_in) * W_in + w_in;
                        int weight_idx = (((c_out * (C_in / groups) + (c_in - c_in_start)) * K_h + k_h) * K_w) + k_w;
                        value += input[input_idx] * weight[weight_idx];
                    }
                }
            }
        }
    }

    int output_idx = ((n * C_out + c_out) * H_out + h_out) * W_out + w_out;
    output[output_idx] = value;
}

// C++ interface to the PyTorch module
torch::Tensor conv2d_cuda_coalesced(
    torch::Tensor input,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias_opt,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> dilation,
    int64_t groups
) {
    // Ensure inputs are contiguous and on CUDA
    input = input.contiguous();
    weight = weight.contiguous();

    TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA");

    if (bias_opt.has_value()) {
        TORCH_CHECK(bias_opt.value().is_cuda(), "Bias tensor must be on CUDA if provided");
    }

    // Input dimensions
    int64_t N = input.size(0);
    int64_t C_in = input.size(1);
    int64_t H_in = input.size(2);
    int64_t W_in = input.size(3);

    // Weight dimensions
    int64_t C_out = weight.size(0);
    int64_t K_h = weight.size(2);
    int64_t K_w = weight.size(3);

    // Stride, padding, and dilation settings
    int64_t stride_h = stride[0];
    int64_t stride_w = stride[1];
    int64_t padding_h = padding[0];
    int64_t padding_w = padding[1];
    int64_t dilation_h = dilation[0];
    int64_t dilation_w = dilation[1];

    // Calculate output dimensions
    int64_t H_out = (H_in + 2 * padding_h - dilation_h * (K_h - 1) - 1) / stride_h + 1;
    int64_t W_out = (W_in + 2 * padding_w - dilation_w * (K_w - 1) - 1) / stride_w + 1;

    // Allocate output tensor
    auto output = torch::zeros({N, C_out, H_out, W_out}, input.options());

    const float* input_ptr = input.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = nullptr;
    torch::Tensor bias;
    if (bias_opt.has_value()) {
        bias = bias_opt.value().contiguous();
        bias_ptr = bias.data_ptr<float>();
    }

    float* output_ptr = output.data_ptr<float>();

    // Launch the optimized kernel with memory coalescing
    dim3 threads_per_block(W_out, H_out);
    dim3 num_blocks(N, C_out);

    conv2d_cuda_kernel_coalesced<<<num_blocks, threads_per_block>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        N, C_in, H_in, W_in,
        C_out, H_out, W_out,
        K_h, K_w,
        stride_h, stride_w,
        padding_h, padding_w,
        dilation_h, dilation_w,
        groups
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in conv2d_cuda_kernel_coalesced: %s\n", hipGetErrorString(err));
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv2d_cuda_coalesced, "Optimized 2D convolution with memory coalescing (CUDA)",
        py::arg("input"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = std::vector<int64_t>{1, 1},
        py::arg("padding") = std::vector<int64_t>{0, 0},
        py::arg("dilation") = std::vector<int64_t>{1, 1},
        py::arg("groups") = 1
    );
}