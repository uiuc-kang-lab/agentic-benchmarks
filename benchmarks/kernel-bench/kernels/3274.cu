#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Improved kernel with optimized indexing for better performance
__global__ void swish_optimized_indexing_kernel(const float* __restrict__ x, float* __restrict__ y, int64_t n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    for (int i = idx; i < n; i += stride) {
        float val = x[i];
        float sigmoid = 1.0f / (1.0f + expf(-val));
        y[i] = val * sigmoid;
    }
}

torch::Tensor swish_optimized_indexing_forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    auto y = torch::empty_like(x);
    const int64_t n = x.numel();
    
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;
    
    swish_optimized_indexing_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        n
    );
    
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &swish_optimized_indexing_forward, "Swish activation forward pass with optimized indexing (CUDA)");
}