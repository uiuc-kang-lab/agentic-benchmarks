#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Device helper: define an inline exponential function for float and double.
template <typename scalar_t>
__device__ inline scalar_t my_exp(scalar_t x);

template <>
__device__ inline float my_exp<float>(float x) {
    return expf(x);
}

template <>
__device__ inline double my_exp<double>(double x) {
    return exp(x);
}

// CUDA kernel implementing the SELU activation with optimized thread and block indexing.
template <typename scalar_t>
__global__ void selu_kernel(const scalar_t* __restrict__ input,
                            scalar_t* __restrict__ output,
                            size_t numel) {
    // Calculate global thread index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;
    
    // SELU parameters
    const scalar_t alpha = static_cast<scalar_t>(1.67326324235437728481);
    const scalar_t lambda = static_cast<scalar_t>(1.05070098735548049342);

    // Grid-stride loop for efficient memory access
    for (; idx < numel; idx += stride) {
        scalar_t x = input[idx];
        scalar_t result = (x > static_cast<scalar_t>(0)) 
                          ? x 
                          : alpha * (my_exp(x) - static_cast<scalar_t>(1));
        output[idx] = lambda * result;
    }
}

// Host function that launches the CUDA SELU kernel.
torch::Tensor selu_forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    auto output = torch::empty_like(input);
    const size_t numel = input.numel();
    const int threads = 512;  // Optimal thread count per block
    const int blocks = (numel + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "selu_forward_cuda", ([&] {
        const scalar_t *input_ptr = input.data_ptr<scalar_t>();
        scalar_t *output_ptr = output.data_ptr<scalar_t>();
        selu_kernel<scalar_t><<<blocks, threads>>>(input_ptr, output_ptr, numel);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &selu_forward, "SELU Activation Forward (CUDA)");
}