#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device function for calculating KL divergence for a single element
__device__ float compute_kl_div(float log_pred, float target) {
    return expf(log_pred) - target * log_pred;
}

// Device function for warp reduction
__device__ float warp_reduce_sum(float val) {
    for (int offset = 16; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__global__ void modular_kl_div_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    const unsigned int warp_size = 32;
    const unsigned int lane_id = threadIdx.x % warp_size;
    const unsigned int warp_id = threadIdx.x / warp_size;
    const unsigned int warps_per_block = blockDim.x / warp_size;
    
    extern __shared__ float warp_results[];
    
    float sum = 0.0f;
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    for (int i = tid; i < n; i += stride) {
        sum += compute_kl_div(log_predictions[i], targets[i]);
    }
    
    sum = warp_reduce_sum(sum);
    
    if (lane_id == 0) {
        warp_results[warp_id] = sum;
    }
    
    __syncthreads();
    
    if (warp_id == 0) {
        float warp_sum = (lane_id < warps_per_block) ? warp_results[lane_id] : 0.0f;
        warp_sum = warp_reduce_sum(warp_sum);
        
        if (lane_id == 0) {
            atomicAdd(output, warp_sum);
        }
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    const int threads_per_block = 256;
    const int num_warps = threads_per_block / 32;
    const int blocks = min(256, (n + threads_per_block - 1) / threads_per_block);
    const int shared_mem = num_warps * sizeof(float);
    
    modular_kl_div_kernel<<<blocks, threads_per_block, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "Modular KL divergence forward (CUDA)");
}
