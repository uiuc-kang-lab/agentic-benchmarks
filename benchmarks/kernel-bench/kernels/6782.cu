#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define BLOCK_DIM_X 32
#define BLOCK_DIM_Y 32

__global__ void prod_reduce_kernel(const float* __restrict__ input, 
                                 float* __restrict__ output,
                                 int stride, int num_elements) {
    // 2D block configuration for better hardware utilization
    const int tid = threadIdx.x + threadIdx.y * BLOCK_DIM_X;
    const int block_size = BLOCK_DIM_X * BLOCK_DIM_Y;
    const int idx = blockIdx.x * block_size + tid;
    
    if (idx < num_elements) {
        float product = 1.0f;
        const float* input_offset = input + idx;
        
        #pragma unroll 10
        for (int i = 0; i < 50; i += 10) {
            product *= input_offset[i * stride];
            product *= input_offset[(i + 1) * stride];
            product *= input_offset[(i + 2) * stride];
            product *= input_offset[(i + 3) * stride];
            product *= input_offset[(i + 4) * stride];
            product *= input_offset[(i + 5) * stride];
            product *= input_offset[(i + 6) * stride];
            product *= input_offset[(i + 7) * stride];
            product *= input_offset[(i + 8) * stride];
            product *= input_offset[(i + 9) * stride];
        }
        
        output[idx] = product;
    }
}

torch::Tensor forward(torch::Tensor x, int dim) {
    CHECK_INPUT(x);

    auto sizes = x.sizes().vec();
    int dim_size = sizes[dim];
    sizes.erase(sizes.begin() + dim);
    torch::Tensor output = torch::empty(sizes, x.options());

    int num_elements = output.numel();
    int stride = x.stride(dim);

    const float* input_ptr = x.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();

    dim3 threads(BLOCK_DIM_X, BLOCK_DIM_Y);
    int blocks = (num_elements + (BLOCK_DIM_X * BLOCK_DIM_Y) - 1) / (BLOCK_DIM_X * BLOCK_DIM_Y);
    
    prod_reduce_kernel<<<blocks, threads>>>(input_ptr, output_ptr, stride, num_elements);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Product reduction over a dimension (CUDA)");
}