#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void conv1d_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int B,
    int in_channels,
    int in_size,
    int out_channels,
    int kernel_size,
    int out_size,
    int stride,
    int dilation
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = B * out_channels * out_size;
    if (idx >= total_elements) return;

    // Calculate indices with minimal divergence
    int o = idx % out_size;
    int tmp = idx / out_size;
    int oc = tmp % out_channels;
    int b = tmp / out_channels;

    // Pre-calculate start and end positions for the convolution window
    int start_pos = o * stride;
    int end_pos = start_pos + (kernel_size - 1) * dilation;
    
    // Skip computation if entire window is out of bounds
    float sum = 0.0f;
    if (end_pos < in_size) {
        // Main convolution loop - no boundary checks needed
        for (int ic = 0; ic < in_channels; ++ic) {
            const float* x_base = x + b * (in_channels * in_size) + ic * in_size + start_pos;
            const float* w_base = weight + oc * (in_channels * kernel_size) + ic * kernel_size;
            
            // Unroll small kernel sizes for better instruction scheduling
            #pragma unroll 4
            for (int k = 0; k < kernel_size; ++k) {
                sum += x_base[k * dilation] * w_base[k];
            }
        }
    } else {
        // Handle boundary case uniformly for the entire warp
        for (int ic = 0; ic < in_channels; ++ic) {
            const float* x_base = x + b * (in_channels * in_size) + ic * in_size;
            const float* w_base = weight + oc * (in_channels * kernel_size) + ic * kernel_size;
            
            #pragma unroll 4
            for (int k = 0; k < kernel_size; ++k) {
                int input_pos = start_pos + k * dilation;
                // Use multiplication instead of branching
                bool valid = input_pos < in_size;
                sum += valid * x_base[input_pos] * w_base[k];
            }
        }
    }

    // Uniform bias addition across warp
    if (bias != nullptr) {
        sum += bias[oc];
    }
    
    output[b * (out_channels * out_size) + oc * out_size + o] = sum;
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int dilation
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(weight.dim() == 3, "weight must be 3D");
    TORCH_CHECK(weight.size(1) == x.size(1), "Input channels mismatch");

    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        TORCH_CHECK(bias->dim() == 1, "bias must be 1D");
        TORCH_CHECK(bias->size(0) == weight.size(0), "Bias size mismatch");
    }

    int B = x.size(0);
    int in_channels = x.size(1);
    int in_size = x.size(2);
    int out_channels = weight.size(0);
    int kernel_size = weight.size(2);

    int out_size = (in_size - dilation * (kernel_size - 1) - 1) / stride + 1;
    TORCH_CHECK(out_size > 0, "Invalid output size");

    auto output = torch::empty({B, out_channels, out_size}, x.options());
    if (output.numel() == 0) return output;

    const float* x_data = x.data_ptr<float>();
    const float* weight_data = weight.data_ptr<float>();
    const float* bias_data = bias ? bias->data_ptr<float>() : nullptr;
    float* output_data = output.data_ptr<float>();

    // Ensure block size is warp-aligned
    int threads = 256;
    int blocks = (B * out_channels * out_size + threads - 1) / threads;

    conv1d_kernel<<<blocks, threads>>>(
        x_data,
        weight_data,
        bias_data,
        output_data,
        B,
        in_channels,
        in_size,
        out_channels,
        kernel_size,
        out_size,
        stride,
        dilation
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel launch error: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "1D convolution forward (CUDA) with uniform warp execution");
}