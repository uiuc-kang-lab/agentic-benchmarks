#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Align memory accesses to 128-bit boundaries
constexpr int ALIGNMENT = 16;

template <typename scalar_t>
__global__ void rms_norm_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int num_features,
    const int numel_per_batch,
    const float eps
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int total_elements = batch_size * numel_per_batch;
    
    for (int idx = tid; idx < total_elements; idx += stride) {
        const int batch_id = idx / numel_per_batch;
        const int offset_in_batch = idx % numel_per_batch;
        const int batch_offset = batch_id * num_features * numel_per_batch;
        
        // Calculate sum of squares using grid-stride loop
        scalar_t sumsq = 0.0f;
        #pragma unroll
        for (int feat = 0; feat < num_features; feat++) {
            const scalar_t val = __ldg(&input[batch_offset + feat * numel_per_batch + offset_in_batch]);
            sumsq += val * val;
        }
        
        // Calculate RMS
        const scalar_t rms = sqrt(sumsq / num_features + eps);
        
        // Normalize using grid-stride loop
        #pragma unroll
        for (int feat = 0; feat < num_features; feat++) {
            const int out_idx = batch_offset + feat * numel_per_batch + offset_in_batch;
            output[out_idx] = input[out_idx] / rms;
        }
    }
}

torch::Tensor rms_norm_cuda_forward(torch::Tensor input, float eps) {
    auto output = torch::empty_like(input);
    
    const int batch_size = input.size(0);
    const int num_features = input.size(1);
    
    int numel_per_batch = 1;
    for(int i = 2; i < input.dim(); i++) {
        numel_per_batch *= input.size(i);
    }

    const int threads_per_block = 512;
    const int max_blocks = 65535;
    const int total_elements = batch_size * numel_per_batch;
    const int blocks = min(max_blocks, (total_elements + threads_per_block - 1) / threads_per_block);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "rms_norm_cuda", ([&] {
        rms_norm_kernel<scalar_t><<<blocks, threads_per_block>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            num_features,
            numel_per_batch,
            eps
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &rms_norm_cuda_forward, "RMS normalization forward (CUDA)");
}