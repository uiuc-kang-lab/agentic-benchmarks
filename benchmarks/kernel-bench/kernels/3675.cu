#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Modular device function for HardSigmoid activation
// Computes y = clamp((x + 3) / 6, 0, 1)
template <typename scalar_t>
__device__ __forceinline__ scalar_t hard_sigmoid(scalar_t x) {
    scalar_t y = (x + static_cast<scalar_t>(3)) * static_cast<scalar_t>(0.16666667);
    return (y < static_cast<scalar_t>(0)) ? static_cast<scalar_t>(0) : ((y > static_cast<scalar_t>(1)) ? static_cast<scalar_t>(1) : y);
}

// CUDA kernel: processes the input tensor using the modular hard_sigmoid device function
template <typename scalar_t>
__global__ void hardsigmoid_kernel(const scalar_t* __restrict__ input,
                                    scalar_t* __restrict__ output,
                                    size_t numel) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = blockDim.x * gridDim.x;
  for (size_t i = idx; i < numel; i += stride) {
    output[i] = hard_sigmoid(input[i]);
  }
}

// Host function to launch the CUDA kernel
torch::Tensor forward(torch::Tensor input) {
  TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");
  auto output = torch::empty_like(input);
  const size_t numel = input.numel();
  const int threads = 1024;
  const int blocks = (numel + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "hardsigmoid_cuda", ([&] {
    hardsigmoid_kernel<scalar_t><<<blocks, threads>>>(
        input.data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>(),
        numel);
  }));

  hipError_t err = hipGetLastError();
  TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));
  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "HardSigmoid activation forward (CUDA)");
}
