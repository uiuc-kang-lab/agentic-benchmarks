#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

__global__ void vectorizedMultiplyKernel(const float* __restrict__ A,
                                        float* __restrict__ C,
                                        float s,
                                        int64_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx4 = idx * 4;
    
    // Process 4 elements at a time using float4
    if (idx4 + 3 < size) {
        float4 a4;
        float4* a4_ptr = (float4*)(&A[idx4]);
        float4* c4_ptr = (float4*)(&C[idx4]);
        
        a4 = *a4_ptr;
        a4.x = __ldg(&A[idx4]) * s;
        a4.y = __ldg(&A[idx4 + 1]) * s;
        a4.z = __ldg(&A[idx4 + 2]) * s;
        a4.w = __ldg(&A[idx4 + 3]) * s;
        
        *c4_ptr = a4;
    }
    // Handle remaining elements
    else if (idx < (size + 3) / 4) {
        int base = idx4;
        for (int i = 0; i < 4 && base + i < size; i++) {
            C[base + i] = __ldg(&A[base + i]) * s;
        }
    }
}

torch::Tensor forward(torch::Tensor A, float s)
{
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor.");
    TORCH_CHECK(A.scalar_type() == torch::kFloat, "Input tensor A must be of type float.");
    
    auto C = torch::empty_like(A);
    int64_t size = A.numel();
    
    const int threads = 256;
    const int blocks = ((size + 3) / 4 + threads - 1) / threads;
    
    vectorizedMultiplyKernel<<<blocks, threads>>>(A.data_ptr<float>(),
                                                 C.data_ptr<float>(),
                                                 s,
                                                 size);
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Vectorized matrix-scalar multiplication kernel");
}