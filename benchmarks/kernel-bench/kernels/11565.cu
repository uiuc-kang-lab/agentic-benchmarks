#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel for KL divergence calculation with warp-level optimization
__global__ void kl_div_kernel_warp_optimized(
    const float* log_predictions,
    const float* targets, 
    float* output,
    const int n) {
    
    // Get global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Shared memory for partial sums
    extern __shared__ float partial_sums[];
    
    float sum = 0.0f;
    
    // Calculate KL divergence for this thread's elements
    for (int i = idx; i < n; i += blockDim.x * gridDim.x) {
        // F.kl_div implementation:
        // output = exp(log_predictions) - targets * log_predictions
        float log_pred = log_predictions[i];
        float target = targets[i];
        sum += expf(log_pred) - target * log_pred;
    }
    
    // Store in shared memory
    partial_sums[threadIdx.x] = sum;
    __syncthreads();
    
    // Parallel reduction in shared memory
    for (int stride = blockDim.x/2; stride > 32; stride >>= 1) {
        if (threadIdx.x < stride) {
            partial_sums[threadIdx.x] += partial_sums[threadIdx.x + stride];
        }
        __syncthreads();
    }
    
    // Warp-level reduction for final 32 elements
    if (threadIdx.x < 32) {
        volatile float* smem = partial_sums;
        smem[threadIdx.x] += smem[threadIdx.x + 32];
        smem[threadIdx.x] += smem[threadIdx.x + 16];
        smem[threadIdx.x] += smem[threadIdx.x + 8];
        smem[threadIdx.x] += smem[threadIdx.x + 4];
        smem[threadIdx.x] += smem[threadIdx.x + 2];
        smem[threadIdx.x] += smem[threadIdx.x + 1];
    }
    
    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        atomicAdd(output, partial_sums[0]);
    }
}

torch::Tensor kl_div_cuda_forward_warp_optimized(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    // Get tensor sizes
    const int n = log_predictions.numel();
    
    // Create output tensor
    auto output = torch::zeros({1}, log_predictions.options());
    
    // Launch parameters
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;
    const int shared_mem = threads * sizeof(float);
    
    // Launch kernel
    kl_div_kernel_warp_optimized<<<blocks, threads, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward_warp_optimized, "KL divergence forward with warp optimization (CUDA)");
}