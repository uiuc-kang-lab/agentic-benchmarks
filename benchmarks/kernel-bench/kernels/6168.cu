#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Use shared memory to store partial sums
template <typename scalar_t>
__global__ void avg_pool2d_forward_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int N,
    int C,
    int H,
    int W,
    int outH,
    int outW,
    int kernel_size,
    int stride,
    int padding
) {
    extern __shared__ scalar_t shared_data[];
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = N * C * outH * outW;
    if (index >= total) {
        return;
    }

    int w_out = index % outW;
    int h_out = (index / outW) % outH;
    int c = (index / (outW * outH)) % C;
    int n = index / (outW * outH * C);

    int h_start = h_out * stride - padding;
    int w_start = w_out * stride - padding;

    scalar_t sum_val = scalar_t(0);
    for (int i = 0; i < kernel_size; i++) {
        for (int j = 0; j < kernel_size; j++) {
            int h_in = h_start + i;
            int w_in = w_start + j;
            if (h_in >= 0 && h_in < H && w_in >= 0 && w_in < W) {
                sum_val += input[((n * C + c) * H + h_in) * W + w_in];
            }
        }
    }
    shared_data[threadIdx.x] = sum_val;
    __syncthreads(); // Ensure all threads have written their partial sums

    if (threadIdx.x == 0) {
        scalar_t block_sum = scalar_t(0);
        for (int i = 0; i < blockDim.x; i++) {
            block_sum += shared_data[i];
        }
        output[index] = block_sum / static_cast<scalar_t>(kernel_size * kernel_size);
    }
}

torch::Tensor avg_pool2d_forward(
    torch::Tensor x,
    int kernel_size,
    int stride,
    int padding
) {
    TORCH_CHECK(x.dim() == 4, "Input must be a 4D tensor.");
    auto N = x.size(0);
    auto C = x.size(1);
    auto H = x.size(2);
    auto W = x.size(3);

    int outH = (H + 2 * padding - kernel_size) / stride + 1;
    int outW = (W + 2 * padding - kernel_size) / stride + 1;

    auto x_cont = x.contiguous();
    auto options = x.options();
    auto out = torch::empty({N, C, outH, outW}, options);

    const int threads = 128;
    const int blocks = (N * C * outH * outW + threads - 1) / threads;
    size_t shared_memory_size = threads * sizeof(scalar_t);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "avg_pool2d_forward_kernel", ([&] {
        const scalar_t* input_data = x_cont.data_ptr<scalar_t>();
        scalar_t* output_data = out.data_ptr<scalar_t>();

        avg_pool2d_forward_kernel<<<blocks, threads, shared_memory_size>>>(
            input_data,
            output_data,
            N, C, H, W,
            outH, outW,
            kernel_size, stride, padding
        );
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err));

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &avg_pool2d_forward, "2D Average Pooling forward (CUDA)");
}