#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    // 2D thread block for better spatial locality
    const int oh = blockIdx.y * blockDim.y + threadIdx.y;
    const int ow = blockIdx.x * blockDim.x + threadIdx.x;
    const int c = blockIdx.z % channels;
    const int b = blockIdx.z / channels;

    if (oh >= output_height || ow >= output_width) return;

    // Calculate input window boundaries
    const int ih_start = oh * stride - padding;
    const int iw_start = ow * stride - padding;

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    // Pre-calculate batch and channel offsets
    const int batch_offset = b * channels * input_height * input_width;
    const int channel_offset = c * input_height * input_width;

    #pragma unroll
    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = ih_start + kh * dilation;
        if (ih >= 0 && ih < input_height) {
            const int row_offset = ih * input_width;
            
            #pragma unroll
            for (int kw = 0; kw < kernel_size; kw++) {
                const int iw = iw_start + kw * dilation;
                if (iw >= 0 && iw < input_width) {
                    const int input_idx = batch_offset + channel_offset + row_offset + iw;
                    max_val = max(max_val, input[input_idx]);
                }
            }
        }
    }

    // Calculate output index using 2D thread configuration
    const int output_idx = b * (channels * output_height * output_width) +
                          c * (output_height * output_width) +
                          oh * output_width +
                          ow;
    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    // Use 2D thread blocks (16x16) for better spatial locality
    const dim3 threads(16, 16);
    const dim3 blocks(
        (output_width + threads.x - 1) / threads.x,
        (output_height + threads.y - 1) / threads.y,
        batch_size * channels
    );

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}