#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

struct PoolConfig {
    int N, C, H, W, outH, outW;
    int kernel_size, stride, padding;
};

__constant__ PoolConfig cfg;

template <typename scalar_t>
__global__ void avg_pool2d_forward_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output
) {
    int nc = blockIdx.z;
    int n = nc / cfg.C;
    int c = nc % cfg.C;
    
    int h_out = blockIdx.x * blockDim.y + threadIdx.y;
    int w_out = blockIdx.y * blockDim.x + threadIdx.x;
    
    if (n >= cfg.N || c >= cfg.C || h_out >= cfg.outH || w_out >= cfg.outW) return;

    int h_start = h_out * cfg.stride - cfg.padding;
    int w_start = w_out * cfg.stride - cfg.padding;

    scalar_t sum_val = scalar_t(0);
    for (int i = 0; i < cfg.kernel_size; ++i) {
        for (int j = 0; j < cfg.kernel_size; ++j) {
            int h_in = h_start + i;
            int w_in = w_start + j;
            if (h_in >= 0 && h_in < cfg.H && w_in >= 0 && w_in < cfg.W) {
                sum_val += input[((n * cfg.C + c) * cfg.H + h_in) * cfg.W + w_in];
            }
        }
    }
    output[((n * cfg.C + c) * cfg.outH + h_out) * cfg.outW + w_out] = 
        sum_val / static_cast<scalar_t>(cfg.kernel_size * cfg.kernel_size);
}

torch::Tensor avg_pool2d_forward(
    torch::Tensor x,
    int kernel_size,
    int stride,
    int padding
) {
    TORCH_CHECK(x.dim() == 4, "Input must be a 4D tensor.");
    auto N = x.size(0);
    auto C = x.size(1);
    auto H = x.size(2);
    auto W = x.size(3);

    int outH = (H + 2 * padding - kernel_size)/stride + 1;
    int outW = (W + 2 * padding - kernel_size)/stride + 1;

    auto x_cont = x.contiguous();
    auto out = torch::empty({N, C, outH, outW}, x.options());

    PoolConfig host_cfg{N, C, H, W, outH, outW, kernel_size, stride, padding};
    hipMemcpyToSymbol(HIP_SYMBOL(cfg), &host_cfg, sizeof(PoolConfig));

    dim3 block(32, 4);
    dim3 grid(
        (outH + block.y - 1) / block.y,
        (outW + block.x - 1) / block.x,
        N * C
    );

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "avg_pool_forward", ([&] {
        avg_pool2d_forward_kernel<scalar_t><<<grid, block>>>(
            x_cont.data_ptr<scalar_t>(),
            out.data_ptr<scalar_t>()
        );
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err));

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &avg_pool2d_forward, "2D Average Pooling forward (CUDA)");
}