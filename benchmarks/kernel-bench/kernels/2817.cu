#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

const int THREADS = 256;
const int ELEMENTS_PER_THREAD = 4;

// Kernel to distribute workload evenly

template <typename scalar_t>
__global__ void sigmoid_kernel(const scalar_t* __restrict__ input,
                             scalar_t* __restrict__ output,
                             const int64_t size) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;

    for (int idx = global_idx; idx < size; idx += total_threads) {
        float val = static_cast<float>(input[idx]);
        output[idx] = static_cast<scalar_t>(1.0f / (1.0f + expf(-val)));
    }
}

torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int64_t size = input.numel();
    
    const int blocks = (size + THREADS - 1) / THREADS;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sigmoid_kernel", [&] {
        const auto* input_data = input.data_ptr<scalar_t>();
        auto* output_data = output.data_ptr<scalar_t>();
        
        sigmoid_kernel<scalar_t><<<blocks, THREADS>>>(input_data, output_data, size);
    });
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Sigmoid forward (CUDA)");
}