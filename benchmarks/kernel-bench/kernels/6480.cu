#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__device__ scalar_t warp_reduce_sum(scalar_t val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

template <typename scalar_t>
__global__ void mean_reduce_kernel(
    const scalar_t* input,
    scalar_t* output,
    int64_t outer_size,
    int64_t dim_size, 
    int64_t inner_size) {

    const int warpSize = 32;
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int warp_id = tid / warpSize;
    const int lane = tid % warpSize;
    if (warp_id >= outer_size * inner_size) return;

    const int outer_idx = warp_id / inner_size;
    const int inner_idx = warp_id % inner_size;
    const int input_offset = outer_idx * dim_size * inner_size + inner_idx;

    scalar_t sum = 0;
    // Each lane processes a subset of the reduction dimension
    for (int i = lane; i < dim_size; i += warpSize) {
        sum += input[input_offset + i * inner_size];
    }

    sum = warp_reduce_sum(sum);
    if (lane == 0) {
        output[warp_id] = sum / dim_size;
    }
}

torch::Tensor mean_reduce_cuda(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();
    
    auto sizes = input.sizes().vec();
    int64_t dim_size = sizes[dim];
    
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }
    
    sizes.erase(sizes.begin() + dim);
    auto output = torch::empty(sizes, input.options());
    
    const int threads = 256;
    const int blocks = (outer_size * inner_size + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES(input.type(), "mean_reduce_cuda", ([&] {
        mean_reduce_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            dim_size,
            inner_size
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &mean_reduce_cuda, "Mean reduction (CUDA)");
}