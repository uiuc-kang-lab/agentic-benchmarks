#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Custom kernel for transposed 1D convolution using optimized thread indexing.
__global__ void conv_transposed_1d_kernel(
    const float* __restrict__ input,   // [N, in_channels, input_width]
    const float* __restrict__ weight,  // [in_channels, out_channels, kernel_size]
    const float* __restrict__ bias,    // [out_channels] or nullptr
    float* __restrict__ output,        // [N, out_channels, output_width]
    int N,
    int in_channels,
    int out_channels,
    int input_width,
    int output_width,
    int kernel_size,
    int stride,
    int padding,
    int output_padding,
    int groups
) {
    // Map threads in a 3D grid: x -> spatial (output width), y -> output channel, z -> batch
    int ox = blockIdx.x * blockDim.x + threadIdx.x;
    if (ox >= output_width) return;
    int oc = blockIdx.y;
    int n  = blockIdx.z;

    // Determine group and channel ranges
    int out_channels_per_group = out_channels / groups;
    int in_channels_per_group = in_channels / groups;
    int group = oc / out_channels_per_group;
    int ic_start = group * in_channels_per_group;

    // Initialize accumulator with bias if provided
    float sum = (bias != nullptr) ? bias[oc] : 0.0f;

    // Each thread computes one output element at (n, oc, ox)
    // Relationship: ox = i * stride - padding + k  =>  i = (ox + padding - k) / stride, if divisible
    for (int ic = 0; ic < in_channels_per_group; ic++) {
        int global_ic = ic_start + ic;
        for (int k = 0; k < kernel_size; k++) {
            int temp = ox + padding - k;
            if (temp < 0) continue;
            if ((temp % stride) != 0) continue;
            int ix = temp / stride;
            if (ix < 0 || ix >= input_width) continue;
            
            // Compute flat indices for input and weight
            int input_index = n * (in_channels * input_width) + global_ic * input_width + ix;
            int weight_index = global_ic * (out_channels * kernel_size) + oc * kernel_size + k;
            sum += input[input_index] * weight[weight_index];
        }
    }
    
    // Write the computed value to the output tensor
    int output_index = n * (out_channels * output_width) + oc * output_width + ox;
    output[output_index] = sum;
}

// Host function interfaced via pybind11 that prepares inputs and launches the kernel
torch::Tensor forward(
    const torch::Tensor& input,
    const torch::Tensor& weight,
    const c10::optional<torch::Tensor>& bias,
    int64_t stride,
    int64_t padding,
    int64_t output_padding,
    int64_t groups
) {
    CHECK_INPUT(input);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    // Extract dimensions from input and weight tensors
    auto input_sizes = input.sizes();  // [N, in_channels, input_width]
    int N = input_sizes[0];
    int in_channels = input_sizes[1];
    int input_width = input_sizes[2];
    
    auto weight_sizes = weight.sizes(); // [in_channels, out_channels, kernel_size]
    int out_channels = weight_sizes[1];
    int kernel_size = weight_sizes[2];

    // Compute the output width as per conv_transpose1d formula
    int output_width = (input_width - 1) * stride - 2 * padding + kernel_size + output_padding;
    
    auto output = torch::zeros({N, out_channels, output_width}, input.options());
    
    // Setup launch configuration with a 3D grid: (output spatial dimension, output channels, batch)
    const int threads = (output_width < 256) ? output_width : 256;
    const int blocks_x = (output_width + threads - 1) / threads;
    dim3 blocks(blocks_x, out_channels, N);
    
    // Get the current CUDA stream
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    conv_transposed_1d_kernel<<<blocks, threads, 0, stream>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.has_value() ? bias.value().data_ptr<float>() : nullptr,
        output.data_ptr<float>(),
        N,
        in_channels,
        out_channels,
        input_width,
        output_width,
        kernel_size,
        stride,
        padding,
        output_padding,
        groups
    );
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Transposed 1D convolution forward (CUDA)");
}
