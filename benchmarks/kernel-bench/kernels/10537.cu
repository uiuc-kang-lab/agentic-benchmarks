#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define WARP_SIZE 32
#define BLOCK_SIZE 256

__global__ void cumsum_kernel_coalesced(const float* __restrict__ input, 
                                      float* __restrict__ output,
                                      int outer_size, int inner_size, int stride) {
    __shared__ float shared_data[BLOCK_SIZE];
    
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int global_idx = bid * blockDim.x + tid;
    
    // Calculate base indices for coalesced access
    const int outer_idx = global_idx / inner_size;
    const int inner_idx = global_idx % inner_size;
    
    if (outer_idx >= outer_size) return;
    
    // Process each slice with coalesced access pattern using tiling
    float running_sum = 0.0f;
    const int base_idx = outer_idx * stride * inner_size + inner_idx;
    
    // Define tile size for processing chunks
    const int TILE_SIZE = 4;
    
    // Process data in tiles
    for (int tile = 0; tile < stride; tile += TILE_SIZE) {
        // Load tile into shared memory
        #pragma unroll
        for (int t = 0; t < TILE_SIZE && (tile + t) < stride; t++) {
            const int curr_idx = base_idx + (tile + t) * inner_size;
            shared_data[tid * TILE_SIZE + t] = __ldg(&input[curr_idx]);
        }
        __syncthreads();
        
        // Process tile
        #pragma unroll
        for (int t = 0; t < TILE_SIZE && (tile + t) < stride; t++) {
            const int curr_idx = base_idx + (tile + t) * inner_size;
            running_sum += shared_data[tid * TILE_SIZE + t];
            output[curr_idx] = running_sum;
        }
        __syncthreads();
    }
}

torch::Tensor forward(torch::Tensor x, int dim) {
    CHECK_INPUT(x);
    
    auto output = torch::empty_like(x);
    int ndim = x.dim();
    dim = (dim + ndim) % ndim;
    
    int outer_size = 1;
    for (int i = 0; i < dim; ++i) {
        outer_size *= x.size(i);
    }
    
    int inner_size = 1;
    for (int i = dim + 1; i < ndim; ++i) {
        inner_size *= x.size(i);
    }
    
    int stride = x.size(dim);
    
    // Calculate grid and block dimensions for optimal occupancy
    const int total_threads = outer_size * inner_size;
    const int num_blocks = (total_threads + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    cumsum_kernel_coalesced<<<num_blocks, BLOCK_SIZE>>>(
        x.data_ptr<float>(), 
        output.data_ptr<float>(),
        outer_size, inner_size, stride
    );
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Coalesced memory access CUDA cumulative sum");
}