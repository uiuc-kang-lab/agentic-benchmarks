#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel with loop unrolling using #pragma unroll directive
template <typename scalar_t>
__global__ void max_reduce_kernel_unroll(
    const scalar_t* input,
    scalar_t* output,
    const int64_t outer_size,
    const int64_t dim_size,
    const int64_t inner_size
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_elements = outer_size * inner_size;
    if (idx >= total_elements) return;

    const int outer_idx = idx / inner_size;
    const int inner_idx = idx % inner_size;
    const int64_t start_idx = outer_idx * dim_size * inner_size + inner_idx;

    scalar_t max_val = input[start_idx];
    
    // Unroll the reduction loop to reduce loop overhead
    #pragma unroll
    for (int i = 1; i < dim_size; i++) {
        const scalar_t val = input[start_idx + i * inner_size];
        max_val = max(max_val, val);
    }

    output[idx] = max_val;
}

// CUDA forward function
torch::Tensor max_reduce_cuda_forward(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();

    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= input.size(i);
    }

    int64_t inner_size = 1;
    for (int i = dim + 1; i < input.dim(); i++) {
        inner_size *= input.size(i);
    }

    const int64_t dim_size = input.size(dim);

    auto output_sizes = input.sizes().vec();
    output_sizes.erase(output_sizes.begin() + dim);
    auto output = torch::empty(output_sizes, input.options());

    const int threads = 256;
    const int blocks = (outer_size * inner_size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "max_reduce_forward", ([&] {
        max_reduce_kernel_unroll<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            dim_size,
            inner_size
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_reduce_cuda_forward, "Max reduce forward (CUDA)");
}
