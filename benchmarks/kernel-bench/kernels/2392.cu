#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// This kernel computes C = A * B^T using warp-level primitives and loop unrolling.
// Each warp computes one output element C[i, j] using registers and fast __shfl_down_sync reduction.

__global__ void warp_matmul_transposed_kernel_unroll(const float* __restrict__ A,
                                                      const float* __restrict__ B,
                                                      float* __restrict__ C,
                                                      int M, int N, int K) {
    // Define warp size
    const unsigned int warpSize = 32;
    // Get lane index within warp and warp id within block
    unsigned int lane = threadIdx.x; // range [0, 31]
    unsigned int warpId = threadIdx.y; // each block can have several warps in y dimension

    // Map each warp to one output element
    int row = blockIdx.y * blockDim.y + warpId;  // index into A (row)
    int col = blockIdx.x;                          // index into B (treated as row due to transposition)

    if (row < M && col < N) {
        float sum = 0.0f;
        
        // Loop unrolling by a factor of 4: each thread processes 4 elements per iteration.
        int k;
        for (k = lane; k <= K - int(warpSize * 4); k += warpSize * 4) {
            float a0 = A[row * K + k];
            float a1 = A[row * K + k + warpSize];
            float a2 = A[row * K + k + 2 * warpSize];
            float a3 = A[row * K + k + 3 * warpSize];
            
            float b0 = B[col * K + k];
            float b1 = B[col * K + k + warpSize];
            float b2 = B[col * K + k + 2 * warpSize];
            float b3 = B[col * K + k + 3 * warpSize];
            
            sum += a0 * b0 + a1 * b1 + a2 * b2 + a3 * b3;
        }
        // Process any remaining elements
        for (; k < K; k += warpSize) {
            sum += A[row * K + k] * B[col * K + k];
        }

        // Warp-level reduction using __shfl_down_sync
        // All 32 threads in the warp participate
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }
        
        // The first thread in the warp writes the result
        if (lane == 0) {
            C[row * N + col] = sum;
        }
    }
}

// Forward function called from PyTorch
// It launches one warp per output element. 
// Block configuration: blockDim.x = 32 (warp size), blockDim.y = warpsPerBlock (e.g., 8).
// Grid configuration: grid.x covers output columns (N) and grid.y covers groups of output rows.

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.dim() == 2, "A must be 2D");
    TORCH_CHECK(B.dim() == 2, "B must be 2D");
    TORCH_CHECK(A.size(1) == B.size(1), "A and B must have the same K dimension");
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be on CUDA");
    TORCH_CHECK(A.is_contiguous() && B.is_contiguous(), "Inputs must be contiguous");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(0);

    auto C = torch::empty({M, N}, A.options());

    // Configure launch parameters
    const int warpSize = 32;
    const int warpsPerBlock = 8; // adjust this based on occupancy
    dim3 block(warpSize, warpsPerBlock);
    dim3 grid(N, (M + warpsPerBlock - 1) / warpsPerBlock);

    warp_matmul_transposed_kernel_unroll<<<grid, block>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, N, K
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel launch failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Warp-level unrolled matrix multiplication with transposed B (CUDA)");
}
