#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32

// CUDA kernel: each warp processes one row of B, broadcasting the diagonal value from A
__global__ void diag_matmul_kernel_warp_broadcast(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    const int64_t N,
    const int64_t M
) {
    // Compute warp-local lane and global warp index
    int lane = threadIdx.x % WARP_SIZE;
    int warpId = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    int totalWarps = (gridDim.x * blockDim.x) / WARP_SIZE;

    for (int row = warpId; row < N; row += totalWarps) {
        float a_val;
        if (lane == 0) {
            a_val = A[row];
        }
        a_val = __shfl_sync(0xffffffff, a_val, 0);

        for (int col = lane; col < M; col += WARP_SIZE) {
            int idx = row * M + col;
            C[idx] = a_val * B[idx];
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.dim() == 1, "A must be a 1D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == B.size(0), "Dimension mismatch: A.size(0) must match B.size(0)");

    A = A.contiguous();
    B = B.contiguous();

    int64_t N = A.size(0);
    int64_t M = B.size(1);

    auto C = torch::empty({N, M}, B.options());

    // Configure kernel: assign one warp per row
    int threadsPerBlock = 128; // Must be multiple of 32
    int warpsPerBlock = threadsPerBlock / WARP_SIZE;
    int totalWarpsNeeded = N; // one warp per row
    int blocks = (totalWarpsNeeded + warpsPerBlock - 1) / warpsPerBlock;

    diag_matmul_kernel_warp_broadcast<<<blocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N,
        M
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Diagonal matrix multiplication using warp-level broadcast");
}
