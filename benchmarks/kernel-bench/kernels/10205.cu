#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void forward_kernel(
    const float* x,
    const float* weight,
    const float* bias,
    float* output,
    int B,
    int IC,
    int OC,
    int H,
    int W
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = B * OC * H * W;
    int stride = blockDim.x * gridDim.x;

    while (idx < total_elements) {
        // Compute 4D indices from linear index
        int w = idx % W;
        int h = (idx / W) % H;
        int oc = (idx / (W * H)) % OC;
        int b = idx / (W * H * OC);

        float sum = 0.0f;
        // For each input channel, accumulate the result
        for (int ic = 0; ic < IC; ++ic) {
            int x_index = b * IC * H * W + ic * H * W + h * W + w;
            int w_index = oc * IC + ic;
            sum += x[x_index] * weight[w_index];
        }
        
        // Add bias if provided
        output[idx] = bias ? sum + bias[oc] : sum;

        idx += stride;
    }
}


torch::Tensor forward_cuda(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias
) {
    TORCH_CHECK(x.is_cuda() && weight.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(x.dim() == 4, "x must be 4D (NCHW)");
    TORCH_CHECK(weight.dim() == 4, "Weight must be 4D (OC, IC, 1, 1)");
    if (bias) {
        TORCH_CHECK(bias->is_cuda(), "Bias must be CUDA tensor");
        TORCH_CHECK(bias->dim() == 1, "Bias must be 1D");
    }

    const int B = x.size(0);
    const int IC = x.size(1);
    const int H = x.size(2);
    const int W = x.size(3);
    const int OC = weight.size(0);

    TORCH_CHECK(weight.size(1) == IC, "Input/output channel mismatch");
    TORCH_CHECK(weight.size(2) == 1 && weight.size(3) == 1, "Kernel must be 1x1");
    if (bias) {
        TORCH_CHECK(bias->size(0) == OC, "Bias/out channel mismatch");
    }

    // Create output tensor
    auto output = torch::empty({B, OC, H, W}, x.options());

    // Get raw pointers
    const float* x_ptr = x.data_ptr<float>();
    const float* w_ptr = weight.data_ptr<float>();
    const float* b_ptr = bias ? bias->data_ptr<float>() : nullptr;
    float* out_ptr = output.data_ptr<float>();

    const int threads = 256;
    const int total_elements = B * OC * H * W;
    const int blocks = (total_elements + threads - 1) / threads;

    forward_kernel<<<blocks, threads>>>(
        x_ptr, w_ptr, b_ptr, out_ptr,
        B, IC, OC, H, W
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_cuda, "Pointwise 2D convolution forward (CUDA)");
}
