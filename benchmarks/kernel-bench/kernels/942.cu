#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor matmul_cuda(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::empty({M, N}, A.options());

    hipblasHandle_t handle;
    hipblasCreate(&handle); hipblasSetStream(handle, stream);

    hipStream_t stream;
    hipStreamCreate(&stream);
    hipblasSetStream(handle, stream);

    const float alpha = 1.0;
    const float beta = 0.0;

    // Perform GEMM directly using cuBLAS column-major format
    hipblasSgemm(handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                N, M, K,
                &alpha,
                B.data_ptr<float>(), N,    // B is treated as transposed
                A.data_ptr<float>(), K,    // A is treated as transposed
                &beta,
                C.data_ptr<float>(), N);

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    hipblasDestroy(handle);
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_cuda, "cuBLAS Matrix Multiplication with CUDA Streams (CUDA)");
}