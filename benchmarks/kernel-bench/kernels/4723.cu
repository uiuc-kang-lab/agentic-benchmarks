#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>

// CUDA kernel with minimized atomic operation for computing sum of squares
__global__ void compute_norm_kernel_atomic_minimized(const float* input, float* norm_out, int numel) {
    __shared__ float shared_sum[256];
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;
    float sum = 0.0f;

    // Each thread computes partial sum
    while (idx < numel) {
        float val = input[idx];
        sum += val * val;
        idx += blockDim.x * gridDim.x;
    }

    // Write sum to shared memory
    shared_sum[tid] = sum;
    __syncthreads();

    // Reduce within block dynamically
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_sum[tid] += shared_sum[tid + stride];
        }
        __syncthreads();
    }

    // Only the first thread in each block adds result to the output
    if (tid == 0) {
        atomicAdd(norm_out, shared_sum[0]);
    }
}

// Kernel for normalizing the tensor
__global__ void normalize_kernel(const float* input, float* output, float norm, int numel) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numel) {
        output[idx] = input[idx] / norm;
    }
}

// PyTorch interface
torch::Tensor forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be on GPU");
    TORCH_CHECK(input.is_contiguous(), "Input tensor must be contiguous");
    TORCH_CHECK(input.scalar_type() == torch::kFloat32, "Input must be float32");

    auto output = torch::empty_like(input);
    auto norm_tensor = torch::zeros({1}, input.options());

    const float* input_ptr = input.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();
    float* norm_ptr = norm_tensor.data_ptr<float>();
    int numel = input.numel();

    const int threads = 256;
    const int blocks = min(65535, (numel + threads - 1) / threads);

    // Compute sum of squares with atomic operation minimized
    compute_norm_kernel_atomic_minimized<<<blocks, threads>>>(input_ptr, norm_ptr, numel);
    
    // Transfer and calculate norm value
    float norm_val;
    hipMemcpy(&norm_val, norm_ptr, sizeof(float), hipMemcpyDeviceToHost);
    norm_val = sqrt(norm_val);

    // Launch normalization kernel
    normalize_kernel<<<blocks, threads>>>(input_ptr, output_ptr, norm_val, numel);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Frobenius norm normalization");
}