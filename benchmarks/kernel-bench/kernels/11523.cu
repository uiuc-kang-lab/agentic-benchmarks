#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Unrolled loop for better performance
__global__ void unrolled_kl_div_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int num_threads = blockDim.x;
    int idx = bid * num_threads + tid;
    
    extern __shared__ float partial_sums[];
    float sum = 0.0f;

    // Process elements with stride of total threads
    for (int i = idx; i < n; i += gridDim.x * num_threads) {
        float log_pred = log_predictions[i];
        float target = targets[i];
        sum += expf(log_pred) - target * log_pred;
    }
    
    // Store in shared memory
    partial_sums[tid] = sum;
    __syncthreads();
    
    // Unrolled parallel reduction in shared memory
    #pragma unroll
    for (int stride = num_threads / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            partial_sums[tid] += partial_sums[tid + stride];
        }
        __syncthreads();
    }
    
    // Write result for this block to global memory
    if (tid == 0) {
        atomicAdd(output, partial_sums[0]);
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    
    // Create output tensor
    auto output = torch::zeros({1}, log_predictions.options());
    
    // Launch parameters
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;
    const int shared_mem = threads * sizeof(float);
    
    // Launch kernel
    unrolled_kl_div_kernel<<<blocks, threads, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward (CUDA)");
}