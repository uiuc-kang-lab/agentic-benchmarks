#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32
#define TILE_SIZE 32

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    int bx = blockIdx.x * TILE_SIZE;
    int by = blockIdx.y * TILE_SIZE;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by + ty;
    int col = bx + tx;
    
    float sum = 0.0f;
    
    // Only compute for lower triangular part
    if (row >= col && row < N && col < N) {
        // Calculate number of tiles needed
        int num_tiles = (row - col + TILE_SIZE) / TILE_SIZE;
        
        for (int t = 0; t < num_tiles; ++t) {
            int tile_idx = col/TILE_SIZE + t;
            
            // Load tile from A and B into shared memory
            if ((tile_idx * TILE_SIZE + tx) <= row && (by + ty) < N) {
                As[ty][tx] = A[row * N + (tile_idx * TILE_SIZE + tx)];
            } else {
                As[ty][tx] = 0.0f;
            }
            
            if ((tile_idx * TILE_SIZE + ty) < N && col < N) {
                Bs[ty][tx] = B[(tile_idx * TILE_SIZE + ty) * N + col];
            } else {
                Bs[ty][tx] = 0.0f;
            }
            
            __syncthreads();
            
            // Compute partial sum for this tile using vectorized operations
            #pragma unroll 8
            for (int k = 0; k < TILE_SIZE; ++k) {
                if ((tile_idx * TILE_SIZE + k) >= col && (tile_idx * TILE_SIZE + k) <= row) {
                    sum += As[ty][k] * Bs[k][tx];
                }
            }
            
            __syncthreads();
        }
        
        if (row < N && col < N) {
            C[row * N + col] = sum;
        }
    } else if (row < N && col < N) {
        // Set upper triangular part to zero
        C[row * N + col] = 0.0f;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}