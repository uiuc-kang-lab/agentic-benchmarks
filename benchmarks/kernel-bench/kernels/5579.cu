#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

__constant__ int const_params[8];  // [kernel_size, stride, padding, dilation]

template <typename scalar_t>
__device__ __forceinline__ bool check_bounds(
    const int h, const int w,
    const int height, const int width
) {
    return (h >= 0 && h < height && w >= 0 && w < width);
}

template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width
) {
    extern __shared__ scalar_t shared_input[];
    
    const int tid = threadIdx.x;
    const int output_idx = blockIdx.x * blockDim.x + tid;
    
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    const int ow = output_idx % output_width;
    const int oh = (output_idx / output_width) % output_height;
    const int c = (output_idx / (output_width * output_height)) % channels;
    const int b = output_idx / (output_width * output_height * channels);

    const int kernel_size = const_params[0];
    const int stride = const_params[1];
    const int padding = const_params[2];
    const int dilation = const_params[3];

    scalar_t max_val = -__int_as_float(0x7f800000);

    const int h_start = oh * stride - padding;
    const int w_start = ow * stride - padding;
    
    #pragma unroll
    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = h_start + kh * dilation;
        
        #pragma unroll
        for (int kw = 0; kw < kernel_size; kw++) {
            const int iw = w_start + kw * dilation;

            if (check_bounds<scalar_t>(ih, iw, input_height, input_width)) {
                const int input_idx = b * (channels * input_height * input_width) +
                                    c * (input_height * input_width) +
                                    ih * input_width +
                                    iw;
                                    
                max_val = max(max_val, __ldg(&input[input_idx]));
            }
        }
    }

    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int params[8] = {kernel_size, stride, padding, dilation};
    hipMemcpyToSymbol(HIP_SYMBOL(const_params), params, sizeof(int) * 8);

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;
    const int shared_mem_size = (threads + kernel_size - 1) * sizeof(float);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<blocks, threads, shared_mem_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width
        );
    }));

    return output;
}