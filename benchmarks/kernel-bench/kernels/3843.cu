#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

// Device function to compute softplus in a numerically stable way
template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_softplus(scalar_t x) {
    if (x > 20.0) {
        return x;
    } else if (x < -20.0) {
        return exp(x);
    } else {
        return log1p(exp(x));
    }
}

// Kernel using shared memory for reduction
template <typename scalar_t>
__global__ void softplus_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    extern __shared__ scalar_t shared_data[];

    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + tid;
    const int stride = blockDim.x * gridDim.x;

    scalar_t sum = 0;
    for (int i = idx; i < size; i += stride) {
        sum += compute_softplus(input[i]);
    }

    shared_data[tid] = sum;
    __syncthreads();

    // Reduce within block using shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}

// CUDA forward function
torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    const int threads = 256;
    const int blocks = (size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "softplus_forward_cuda", ([&] {
        softplus_kernel<scalar_t><<<blocks, threads, threads * sizeof(scalar_t)>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}