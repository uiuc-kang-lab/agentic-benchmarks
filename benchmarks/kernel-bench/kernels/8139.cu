#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void conv_transpose2d_kernel_shared(
    const scalar_t* input,
    const scalar_t* weight,
    const scalar_t* bias,
    scalar_t* output,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int out_channels,
    const int kernel_h,
    const int kernel_w,
    const int stride,
    const int padding,
    const int output_padding,
    const int groups,
    const int dilation,
    const int out_height,
    const int out_width
) {
    extern __shared__ char shared_memory[];
    scalar_t* shared_weights = reinterpret_cast<scalar_t*>(shared_memory);
    
    const int total_elements = batch_size * out_channels * out_height * out_width;
    const int thread_id = threadIdx.x;
    const int block_size = blockDim.x;
    
    // Calculate weight dimensions
    const int out_channels_per_group = out_channels / groups;
    const int in_channels_per_group = in_channels / groups;
    const int weights_per_group = in_channels_per_group * out_channels_per_group * kernel_h * kernel_w;
    
    // Load weights into shared memory
    for (int i = thread_id; i < weights_per_group; i += block_size) {
        const int ic = (i / (out_channels_per_group * kernel_h * kernel_w)) % in_channels_per_group;
        const int oc = (i / (kernel_h * kernel_w)) % out_channels_per_group;
        const int kh = (i / kernel_w) % kernel_h;
        const int kw = i % kernel_w;
        
        shared_weights[i] = weight[ic * (out_channels_per_group * kernel_h * kernel_w) +
                                 oc * kernel_h * kernel_w +
                                 kh * kernel_w +
                                 kw];
    }
    __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int grid_stride = blockDim.x * gridDim.x;

    for (; idx < total_elements; idx += grid_stride) {
        int n = idx;
        const int ow = n % out_width;
        n /= out_width;
        const int oh = n % out_height;
        n /= out_height;
        const int oc = n % out_channels;
        n /= out_channels;
        const int b = n;

        const int g = oc / out_channels_per_group;
        const int oc_group = oc % out_channels_per_group;
        const int ic_start = g * in_channels_per_group;

        scalar_t val = (bias != nullptr) ? bias[oc] : static_cast<scalar_t>(0);

        #pragma unroll
        for (int kh = 0; kh < kernel_h; ++kh) {
            const int h_in = (oh - kh * dilation + padding) / stride;
            const bool valid_h = (h_in >= 0 && h_in < in_height && 
                                (oh - kh * dilation + padding) % stride == 0);

            if (valid_h) {
                #pragma unroll
                for (int kw = 0; kw < kernel_w; ++kw) {
                    const int w_in = (ow - kw * dilation + padding) / stride;
                    const bool valid_w = (w_in >= 0 && w_in < in_width && 
                                       (ow - kw * dilation + padding) % stride == 0);

                    if (valid_w) {
                        #pragma unroll
                        for (int ic = 0; ic < in_channels_per_group; ++ic) {
                            const scalar_t x_val = input[b * in_channels * in_height * in_width +
                                                      (ic_start + ic) * in_height * in_width +
                                                      h_in * in_width + w_in];

                            const int weight_idx = ic * (out_channels_per_group * kernel_h * kernel_w) +
                                                 oc_group * kernel_h * kernel_w +
                                                 kh * kernel_w + kw;

                            val += x_val * shared_weights[weight_idx];
                        }
                    }
                }
            }
        }
        output[idx] = val;
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int output_padding,
    int groups,
    int dilation = 1
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(x.dim() == 4, "Input must be 4D");
    TORCH_CHECK(weight.dim() == 4, "Weight must be 4D");

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);
    const int out_channels = weight.size(1) * groups;
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);

    if (bias.has_value() && bias->defined()) {
        TORCH_CHECK(bias->numel() == out_channels, "Bias must have out_channels elements");
        TORCH_CHECK(bias->device().is_cuda(), "Bias must be a CUDA tensor");
    }

    const int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_h - 1) + output_padding + 1;
    const int out_width = (in_width - 1) * stride - 2 * padding + dilation * (kernel_w - 1) + output_padding + 1;

    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, x.options());

    const int total_elements = output.numel();
    constexpr int BLOCK_SIZE = 256;
    const int blocks = (total_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Calculate shared memory size
    const int out_channels_per_group = out_channels / groups;
    const int in_channels_per_group = in_channels / groups;
    const size_t shared_memory_size = in_channels_per_group * out_channels_per_group * kernel_h * kernel_w * sizeof(float);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "conv_transpose2d_cuda_shared", ([&] {
        conv_transpose2d_kernel_shared<scalar_t><<<blocks, BLOCK_SIZE, shared_memory_size>>>(
            x.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            (bias.has_value() && bias->defined()) ? bias->data_ptr<scalar_t>() : nullptr,
            output.data_ptr<scalar_t>(),
            batch_size,
            in_channels,
            in_height,
            in_width,
            out_channels,
            kernel_h,
            kernel_w,
            stride,
            padding,
            output_padding,
            groups,
            dilation,
            out_height,
            out_width
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Transposed 2D convolution with shared memory optimization (CUDA)",
          py::arg("x"), py::arg("weight"), py::arg("bias") = py::none(),
          py::arg("stride"), py::arg("padding"), py::arg("output_padding"),
          py::arg("groups"), py::arg("dilation") = 1);
}