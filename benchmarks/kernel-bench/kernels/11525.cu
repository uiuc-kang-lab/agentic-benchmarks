#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device functions for modular kernel components
__device__ __forceinline__ float compute_kl_div(float log_pred, float target) {
    return __expf(log_pred) - target * log_pred;
}

__device__ __forceinline__ float warp_reduce_sum(float val) {
    #pragma unroll
    for (int offset = 32/2; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__device__ __forceinline__ void block_reduce_sum(float* shared_data, const int tid) {
    const int lane_id = tid % 32;
    const int warp_id = tid / 32;
    
    // First warp reduces all partial sums
    if (warp_id == 0) {
        float sum = (tid < blockDim.x/32) ? shared_data[tid] : 0.0f;
        
        // Reduce within warp
        sum = warp_reduce_sum(sum);
        
        // First thread writes result
        if (lane_id == 0) {
            shared_data[0] = sum;
        }
    }
}

__global__ void modular_kl_div_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    const int tid = threadIdx.x;
    const int gid = blockIdx.x * blockDim.x + tid;
    const int stride = gridDim.x * blockDim.x;
    
    // Shared memory for partial sums
    extern __shared__ float shared_mem[];
    
    // Local accumulator
    float thread_sum = 0.0f;
    
    // Process elements with grid stride loop
    for (int idx = gid; idx < n; idx += stride) {
        thread_sum += compute_kl_div(log_predictions[idx], targets[idx]);
    }
    
    // Reduce within warp first
    thread_sum = warp_reduce_sum(thread_sum);
    
    // Write warp results to shared memory
    if (tid % 32 == 0) {
        shared_mem[tid/32] = thread_sum;
    }
    __syncthreads();
    
    // Block-level reduction
    block_reduce_sum(shared_mem, tid);
    
    // First thread adds block result to global output
    if (tid == 0) {
        atomicAdd(output, shared_mem[0]);
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    // Launch parameters
    const int threads = 256;
    const int blocks = min((n + threads - 1) / threads, 1024);
    const int shared_mem = (threads/32) * sizeof(float);
    
    modular_kl_div_kernel<<<blocks, threads, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward (CUDA)");
}