#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t, int KERNEL_SIZE=3>
__global__ void optimized_max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int stride,
    const int padding,
    const int dilation
) {
    __shared__ scalar_t shared_input[32][32];
    
    const int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    const int ow = output_idx % output_width;
    const int oh = (output_idx / output_width) % output_height;
    const int c = (output_idx / (output_width * output_height)) % channels;
    const int b = output_idx / (output_width * output_height * channels);

    scalar_t max_val = -__FLT_MAX__;

    #pragma unroll
    for (int kh = 0; kh < KERNEL_SIZE; kh++) {
        #pragma unroll
        for (int kw = 0; kw < KERNEL_SIZE; kw++) {
            const int ih = oh * stride - padding + kh * dilation;
            const int iw = ow * stride - padding + kw * dilation;

            if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
                const int input_idx = b * (channels * input_height * input_width) +
                                    c * (input_height * input_width) +
                                    ih * input_width +
                                    iw;
                max_val = max(max_val, input[input_idx]);
            }
        }
    }

    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward_optimized(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    const int num_streams = 4;
    std::vector<hipStream_t> streams(num_streams);
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
    }

    const int work_per_stream = blocks / num_streams;
    
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool2d_cuda_forward_optimized", ([&] {
        for (int i = 0; i < num_streams; i++) {
            const int stream_blocks = (i == num_streams - 1) ? blocks - i * work_per_stream : work_per_stream;
            optimized_max_pool2d_kernel<scalar_t><<<stream_blocks, threads, 0, streams[i]>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>() + i * work_per_stream * threads,
                batch_size,
                channels,
                input_height,
                input_width,
                output_height,
                output_width,
                stride,
                padding,
                dilation
            );
        }
    }));

    for (int i = 0; i < num_streams; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward_optimized, "Optimized Max Pool 2D forward (CUDA)");
}