#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32
#define TILE_SIZE 32

__global__ void matmul_reduce_sync_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x, ty = threadIdx.y;
    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;

    float sum = 0.0f;

    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        int k_base = t * TILE_SIZE;

        if (row < M && (k_base + tx) < K) {
            As[ty][tx] = A[row * K + k_base + tx];
        } else {
            As[ty][tx] = 0.0f;
        }

        if (col < N && (k_base + ty) < K) {
            Bs[ty][tx] = B[col * K + k_base + ty];
        } else {
            Bs[ty][tx] = 0.0f;
        }

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[ty][k] * Bs[k][tx];
        }

        if ((k_base + TILE_SIZE) < K) {
            __syncthreads();
        }
    }

    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.dim() == 2, "A must be 2D");
    TORCH_CHECK(B.dim() == 2, "B must be 2D");
    TORCH_CHECK(A.size(1) == B.size(1), "A and B must have same K dimension");
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be on CUDA");
    TORCH_CHECK(A.is_contiguous() && B.is_contiguous(), "Inputs must be contiguous");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(0);

    auto C = torch::empty({M, N}, A.options());
    
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
    dim3 block(TILE_SIZE, TILE_SIZE);
    
    matmul_reduce_sync_kernel<<<grid, block>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, N, K
    );
    
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel failed: ", hipGetErrorString(err));
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication with transposed B optimized for reduced synchronizations (CUDA)");
}