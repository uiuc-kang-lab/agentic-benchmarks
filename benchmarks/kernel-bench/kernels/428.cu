#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32
#define BLOCK_SIZE 256

template <typename scalar_t>
__global__ void matvec_mul_kernel(
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> A,
    const torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> B,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> C,
    int64_t M,
    int64_t K)
{
    __shared__ scalar_t shared_mem[BLOCK_SIZE];
    
    const int64_t row = blockIdx.x;
    const int64_t tid = threadIdx.x;
    const int64_t lane_id = tid % WARP_SIZE;
    
    if (row < M) {
        // Each thread computes partial sum for a segment
        scalar_t thread_sum = 0;
        for (int64_t k = tid; k < K; k += BLOCK_SIZE) {
            thread_sum += A[row][k] * B[k];
        }
        
        // First, perform warp-level reduction within each warp
        for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
            thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
        }
        // Each warp writes its reduced sum to shared memory
        if (lane_id == 0) {
            shared_mem[tid / WARP_SIZE] = thread_sum;
        }
        __syncthreads();
        
        // Let the first warp accumulate the warp sums
        if (tid < BLOCK_SIZE / WARP_SIZE) {
            thread_sum = shared_mem[tid];
            for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
                thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
            }
            if (tid == 0) {
                C[row][0] = thread_sum;
            }
        }
    }
}

torch::Tensor matvec_mul_cuda(torch::Tensor A, torch::Tensor B)
{
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");

    A = A.contiguous();
    B = B.contiguous();

    int64_t M = A.size(0);
    int64_t K = A.size(1);

    TORCH_CHECK(B.numel() == K, "B must have the same number of elements as columns in A");
    TORCH_CHECK(B.dim() == 1 || (B.dim() == 2 && B.size(1) == 1), "B must be a vector");

    auto B_flat = B.view({-1});
    auto C = torch::zeros({M, 1}, A.options());

    dim3 threads(BLOCK_SIZE);
    dim3 blocks(M);

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matvec_mul_cuda", ([&] {
        matvec_mul_kernel<scalar_t><<<blocks, threads>>>(
            A.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
            B_flat.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
            C.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
            M,
            K);
    }));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matvec_mul_cuda, "Matrix-Vector Multiplication (CUDA)");
}