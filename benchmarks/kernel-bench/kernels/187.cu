#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <iostream>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define TILE_SIZE 16

__global__ void matrix_multiply_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    __shared__ float s_A[TILE_SIZE][TILE_SIZE];
    __shared__ float s_B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float value = 0.0f;

    // Loop over tiles
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        int tiledCol = t * TILE_SIZE + threadIdx.x;
        int tiledRow = t * TILE_SIZE + threadIdx.y;
        if (row < M && tiledCol < K)
            s_A[threadIdx.y][threadIdx.x] = A[row * K + tiledCol];
        else
            s_A[threadIdx.y][threadIdx.x] = 0.0f;

        if (tiledRow < K && col < N)
            s_B[threadIdx.y][threadIdx.x] = B[tiledRow * N + col];
        else
            s_B[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            value += s_A[threadIdx.y][k] * s_B[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < M && col < N)
        C[row * N + col] = value;
}

void matrix_multiply_cuda(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
    // Ensure inputs are CUDA tensors and contiguous
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);

    // Get the dimensions of the matrices
    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    // Get the pointers to the data
    const float *d_A = A.data_ptr<float>();
    const float *d_B = B.data_ptr<float>();
    float *d_C = C.data_ptr<float>();

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    matrix_multiply_kernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, K);
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    // Ensure inputs are CUDA tensors and contiguous
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    // Get the dimensions of the matrices
    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    // Create the output tensor
    torch::Tensor C = torch::zeros({M, N}, A.options());

    // Perform the matrix multiplication
    matrix_multiply_cuda(A, B, C);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication with memory coalescing (CUDA)");
}