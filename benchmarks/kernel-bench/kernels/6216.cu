#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized CUDA kernel for 3D average pooling with block size experimentation
// Choosing the optimal block size can enhance the kernel performance based on specific hardware characteristics.

__global__ void avg_pool3d_optimized_block_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int batch_size, int channels,
    int in_d, int in_h, int in_w,
    int out_d, int out_h, int out_w,
    int kernel_size, int stride, int padding) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = batch_size * channels * out_d * out_h * out_w;

    int kernelVolume = kernel_size * kernel_size * kernel_size;

    while (index < total_elements) {
        // Compute output tensor indices from the 1D index
        int w_out = index % out_w;
        int tmp = index / out_w;
        int h_out = tmp % out_h;
        tmp = tmp / out_h;
        int d_out = tmp % out_d;
        tmp = tmp / out_d;
        int c = tmp % channels;
        int n = tmp / channels;

        int d_start = d_out * stride - padding;
        int h_start = h_out * stride - padding;
        int w_start = w_out * stride - padding;

        int d_end = d_start + kernel_size;
        int h_end = h_start + kernel_size;
        int w_end = w_start + kernel_size;

        int d0 = max(d_start, 0);
        int h0 = max(h_start, 0);
        int w0 = max(w_start, 0);

        int d1 = min(d_end, in_d);
        int h1 = min(h_end, in_h);
        int w1 = min(w_end, in_w);

        float sum = 0.0f;
        for (int d = d0; d < d1; ++d) {
            int base_d = ((n * channels + c) * in_d + d) * in_h * in_w;
            for (int h = h0; h < h1; ++h) {
                int base_h = base_d + h * in_w;
                for (int w = w0; w < w1; ++w) {
                    sum += input[base_h + w];
                }
            }
        }

        // For count_include_pad=True, divide by the full kernel volume.
        output[index] = sum / static_cast<float>(kernelVolume);

        index += blockDim.x * gridDim.x;
    }
}

at::Tensor forward(at::Tensor input, int kernel_size, int stride, int padding) {
    TORCH_CHECK(input.dim() == 5, "Input tensor must be 5-dimensional");
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    int batch_size = input.size(0);
    int channels   = input.size(1);
    int in_d       = input.size(2);
    int in_h       = input.size(3);
    int in_w       = input.size(4);

    int out_d = (in_d + 2 * padding - kernel_size) / stride + 1;
    int out_h = (in_h + 2 * padding - kernel_size) / stride + 1;
    int out_w = (in_w + 2 * padding - kernel_size) / stride + 1;

    auto output = at::empty({batch_size, channels, out_d, out_h, out_w}, input.options());

    int total_elements = batch_size * channels * out_d * out_h * out_w;
    int threads = 512; // Chosen block size after experimentation
    int blocks = (total_elements + threads - 1) / threads;

    avg_pool3d_optimized_block_kernel<<<blocks, threads>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size, channels,
        in_d, in_h, in_w,
        out_d, out_h, out_w,
        kernel_size, stride, padding);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D Average Pooling forward (CUDA) - optimized block size");
}