#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cudnn/Handles.h>
#include <ATen/cudnn/Descriptors.h>
#include <hipDNN.h>

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
#define BLOCK_SIZE_Z 4

__global__ void conv3d_minimized_divergence_kernel(
    float* output, const float* input, const float* weight, const float* bias,
    int batch_size, int in_channels, int out_channels,
    int in_depth, int in_height, int in_width,
    int kernel_d, int kernel_h, int kernel_w,
    int out_depth, int out_height, int out_width,
    int stride, int padding, int dilation, int groups) {

    int w_out = blockIdx.x * blockDim.x + threadIdx.x;
    int h_out = blockIdx.y * blockDim.y + threadIdx.y;
    int d_out = blockIdx.z * blockDim.z + threadIdx.z;

    if (w_out >= out_width || h_out >= out_height || d_out >= out_depth) return;

    for (int b = 0; b < batch_size; ++b) {
        for (int c_out = 0; c_out < out_channels; ++c_out) {
            float sum = 0.0f;
            int group = c_out / (out_channels / groups);
            int in_channels_per_group = in_channels / groups;

            for (int ic = 0; ic < in_channels_per_group; ++ic) {
                int in_c = group * in_channels_per_group + ic;

                for (int kd = 0; kd < kernel_d; ++kd) {
                    int d_in = d_out * stride - padding + kd * dilation;
                    bool valid_d = (d_in >= 0 && d_in < in_depth);

                    for (int kh = 0; kh < kernel_h; ++kh) {
                        int h_in = h_out * stride - padding + kh * dilation;
                        bool valid_h = (h_in >= 0 && h_in < in_height);

                        for (int kw = 0; kw < kernel_w; ++kw) {
                            int w_in = w_out * stride - padding + kw * dilation;
                            bool valid_w = (w_in >= 0 && w_in < in_width);

                            if (valid_d && valid_h && valid_w) {
                                int input_index = ((b * in_channels + in_c) * in_depth + d_in) * in_height * in_width +
                                                  h_in * in_width + w_in;
                                int weight_index = (((c_out * in_channels_per_group) + ic) * kernel_d + kd) *
                                                   kernel_h * kernel_w + kh * kernel_w + kw;
                                sum += input[input_index] * weight[weight_index];
                            }
                        }
                    }
                }
            }

            if (bias != nullptr) {
                sum += bias[c_out];
            }

            int output_index = ((b * out_channels + c_out) * out_depth + d_out) * out_height * out_width +
                               h_out * out_width + w_out;
            output[output_index] = sum;
        }
    }
}

at::Tensor forward(
    const at::Tensor& input,
    const at::Tensor& weight,
    const c10::optional<at::Tensor>& bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups
) {
    auto bias = bias_opt.value_or(at::Tensor());
    TORCH_CHECK(input.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "Weight must be a CUDA tensor");
    TORCH_CHECK(!bias.defined() || bias.is_cuda(), "Bias must be a CUDA tensor");

    int64_t batch_size = input.size(0);
    int64_t in_channels = input.size(1);
    int64_t in_depth = input.size(2);
    int64_t in_height = input.size(3);
    int64_t in_width = input.size(4);

    int64_t out_channels = weight.size(0);
    int64_t kernel_d = weight.size(2);
    int64_t kernel_h = weight.size(3);
    int64_t kernel_w = weight.size(4);

    int64_t out_depth = (in_depth + 2 * padding - dilation * (kernel_d - 1) - 1) / stride + 1;
    int64_t out_height = (in_height + 2 * padding - dilation * (kernel_h - 1) - 1) / stride + 1;
    int64_t out_width = (in_width + 2 * padding - dilation * (kernel_w - 1) - 1) / stride + 1;

    auto output = at::empty({batch_size, out_channels, out_depth, out_height, out_width}, input.options());

    dim3 threadsPerBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    dim3 numBlocks(
        (out_width + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X,
        (out_height + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y,
        (out_depth + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z
    );

    conv3d_minimized_divergence_kernel<<<numBlocks, threadsPerBlock>>>(
        output.data_ptr<float>(),
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.defined() ? bias.data_ptr<float>() : nullptr,
        batch_size, in_channels, out_channels,
        in_depth, in_height, in_width,
        kernel_d, kernel_h, kernel_w,
        out_depth, out_height, out_width,
        stride, padding, dilation, groups
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D convolution forward with minimized warp divergence (CUDA)");
}