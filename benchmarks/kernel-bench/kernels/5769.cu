#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>

// This kernel uses a grid-stride loop to evenly distribute the work across threads and blocks.
// Each thread processes multiple output elements if necessary, eliminating load imbalance when the total work is not a multiple of the block size.

template <typename scalar_t>
__global__ void max_pool2d_kernel_even_workload(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int total_elements = batch_size * channels * output_height * output_width;
    
    // Grid-stride loop to distribute workload evenly among threads
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < total_elements; idx += blockDim.x * gridDim.x) {
        const int ow = idx % output_width;
        const int oh = (idx / output_width) % output_height;
        const int c = (idx / (output_width * output_height)) % channels;
        const int b = idx / (output_width * output_height * channels);

        scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

        for (int kh = 0; kh < kernel_size; ++kh) {
            for (int kw = 0; kw < kernel_size; ++kw) {
                const int ih = oh * stride - padding + kh * dilation;
                const int iw = ow * stride - padding + kw * dilation;
                
                if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
                    const int input_idx = b * (channels * input_height * input_width) +
                                          c * (input_height * input_width) +
                                          ih * input_width +
                                          iw;
                    max_val = max(max_val, input[input_idx]);
                }
            }
        }
        output[idx] = max_val;
    }
}

// Forward function to launch the kernel
torch::Tensor max_pool2d_cuda_forward_even_workload(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width  = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int threads = 256;
    const int total_elements = batch_size * channels * output_height * output_width;
    const int blocks = (total_elements + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool2d_cuda_forward_even_workload", ([&] {
        max_pool2d_kernel_even_workload<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward_even_workload, "Max Pool 2D forward with even workload distribution (CUDA)");
}
