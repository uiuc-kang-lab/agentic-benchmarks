#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Constant memory for kernel weights and bias
__constant__ float c_weight[1024];  // Adjust size based on maximum expected kernel weights
__constant__ float c_bias[512];     // Adjust size based on maximum expected bias elements

__global__ void __launch_bounds__(512, 2) depthwise_conv2d_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int input_h,
    int input_w,
    int out_channels,
    int output_h,
    int output_w,
    int kernel_size,
    int stride,
    int padding,
    int channels_per_group,
    bool has_bias
) {
    // Use registers for frequently accessed values
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_elements = batch_size * out_channels * output_h * output_w;
    if (tid >= total_elements) return;

    // Compute output indices more efficiently using bit operations where possible
    const int w_out = tid % output_w;
    const int h_out = (tid / output_w) % output_h;
    const int oc = (tid / (output_w * output_h)) % out_channels;
    const int b = tid / (output_w * output_h * out_channels);

    int in_ch = oc / channels_per_group;
    int weight_ch = oc % channels_per_group;

    float sum = 0.0f;
    
    // Pre-compute base indices
    int input_batch_offset = b * (in_channels * input_h * input_w);
    int input_channel_offset = in_ch * (input_h * input_w);
    int weight_offset = in_ch * (channels_per_group * kernel_size * kernel_size)
                      + weight_ch * (kernel_size * kernel_size);

    #pragma unroll
    for (int kh = 0; kh < kernel_size; ++kh) {
        int h_in = h_out * stride + kh - padding;
        if (h_in >= 0 && h_in < input_h) {
            #pragma unroll
            for (int kw = 0; kw < kernel_size; ++kw) {
                int w_in = w_out * stride + kw - padding;
                if (w_in >= 0 && w_in < input_w) {
                    int input_idx = input_batch_offset
                                  + input_channel_offset
                                  + h_in * input_w
                                  + w_in;
                    int weight_idx = weight_offset
                                   + kh * kernel_size
                                   + kw;

                    sum += __ldg(&input[input_idx]) * c_weight[weight_idx];
                }
            }
        }
    }
    
    if (has_bias) {
        sum += c_bias[oc];
    }
    
    output[b * out_channels * output_h * output_w +
           oc * output_h * output_w +
           h_out * output_w +
           w_out] = sum;
}

torch::Tensor forward(
    torch::Tensor input,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding
) {
    TORCH_CHECK(input.is_cuda() && weight.is_cuda(), "Inputs must be CUDA tensors");
    if (bias.has_value()) {
        TORCH_CHECK(bias->is_cuda(), "Bias must be a CUDA tensor");
    }
    TORCH_CHECK(input.is_contiguous() && weight.is_contiguous(), "Input and weight must be contiguous");
    if (bias.has_value()) {
        TORCH_CHECK(bias->is_contiguous(), "Bias must be contiguous");
    }

    int batch_size = input.size(0);
    int in_channels = input.size(1);
    int input_h = input.size(2);
    int input_w = input.size(3);
    int kernel_size = weight.size(2);
    int channels_per_group = weight.size(1);
    int out_channels = in_channels * channels_per_group;

    if (bias.has_value()) {
        TORCH_CHECK(bias->size(0) == out_channels, "Bias size mismatch");
    }

    int output_h = (input_h + 2 * padding - kernel_size) / stride + 1;
    int output_w = (input_w + 2 * padding - kernel_size) / stride + 1;

    auto output = torch::empty({batch_size, out_channels, output_h, output_w}, input.options());

    // Copy weight to constant memory
    int weight_size = weight.numel() * sizeof(float);
    TORCH_CHECK(weight_size <= 1024 * sizeof(float), "Weight size exceeds constant memory capacity");
    hipMemcpyToSymbol(HIP_SYMBOL(c_weight), weight.data_ptr<float>(), weight_size);

    bool has_bias = bias.has_value();
    if (has_bias) {
        hipMemcpyToSymbol(HIP_SYMBOL(c_bias), bias->data_ptr<float>(), out_channels * sizeof(float));
    }

    int threads = 512;
    int blocks = (batch_size * out_channels * output_h * output_w + threads - 1) / threads;

    depthwise_conv2d_kernel<<<blocks, threads>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        input_h,
        input_w,
        out_channels,
        output_h,
        output_w,
        kernel_size,
        stride,
        padding,
        channels_per_group,
        has_bias
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Depthwise 2D Convolution (CUDA)",
          py::arg("input"), py::arg("weight"), py::arg("bias") = py::none(), py::arg("stride"), py::arg("padding"));
}
