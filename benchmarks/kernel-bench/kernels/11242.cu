#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void smooth_l1_loss_kernel(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    int n_elements
) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int blockSize = blockDim.x;
    const int gid = bid * blockSize + tid;
    const int stride = gridDim.x * blockDim.x * 4;  // Increased stride for unrolling
    
    float thread_sum = 0.0f;
    
    // Manual loop unrolling with 4 elements per iteration
    for (int i = gid * 4; i < n_elements; i += stride) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            if (i + j < n_elements) {
                const float diff = predictions[i + j] - targets[i + j];
                const float abs_diff = fabsf(diff);
                thread_sum += (abs_diff < 1.0f) ? 
                    (0.5f * diff * diff) : 
                    (abs_diff - 0.5f);
            }
        }
    }
    
    __shared__ float shared_sum[256];
    shared_sum[tid] = thread_sum;
    __syncthreads();
    
    // Explicitly unrolled reduction
    if (tid < 128) shared_sum[tid] += shared_sum[tid + 128];
    __syncthreads();
    if (tid < 64) shared_sum[tid] += shared_sum[tid + 64];
    __syncthreads();
    
    // Last warp reduction unrolled
    if (tid < 32) {
        volatile float* smem = shared_sum;
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }
    
    if (tid == 0) {
        atomicAdd(output, shared_sum[0] / n_elements);
    }
}

torch::Tensor smooth_l1_loss_cuda(
    torch::Tensor predictions,
    torch::Tensor targets
) {
    TORCH_CHECK(
        predictions.sizes() == targets.sizes(),
        "Input tensors must have the same shape"
    );
    TORCH_CHECK(
        predictions.is_contiguous() && targets.is_contiguous(),
        "Input tensors must be contiguous"
    );
    TORCH_CHECK(
        predictions.device().is_cuda() && targets.device().is_cuda(),
        "Inputs must be CUDA tensors"
    );

    int n = predictions.numel();
    auto output = torch::zeros({1}, predictions.options());

    const int block_size = 256;
    const int grid_size = (n + block_size * 4 - 1) / (block_size * 4);

    smooth_l1_loss_kernel<<<grid_size, block_size>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &smooth_l1_loss_cuda, "Smooth L1 Loss (CUDA)");
}