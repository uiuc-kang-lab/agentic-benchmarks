#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Optimized Swish kernel using a grid-stride loop to process workloads larger than the number of threads
__global__ void swish_kernel_strided(const float* __restrict__ x, float* __restrict__ y, int64_t n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < n; i += stride) {
        float v = x[i];
        float sig = 1.0f / (1.0f + expf(-v));
        y[i] = v * sig;
    }
}

torch::Tensor swish_forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    auto y = torch::empty_like(x);
    int64_t n = x.numel();
    const int threads = 256;
    int blocks = (n + threads - 1) / threads;

    swish_kernel_strided<<<blocks, threads>>>(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        n
    );

    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &swish_forward, "Swish activation forward pass (CUDA) with grid-stride loop");
}
