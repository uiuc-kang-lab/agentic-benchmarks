#include "hip/hip_runtime.h"
__device__ __forceinline__ float compute_conv_transpose_at_pixel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    int b, int in_channels, int in_height, int in_width,
    int out_channels, int kernel_size,
    int h_out, int w_out, int stride, int padding, int dilation, int o) {
    
  float sum = 0.0f;
  #pragma unroll 4
  for (int c = 0; c < in_channels; ++c) {
    for (int p = 0; p < kernel_size; ++p) {
      int h_unscaled = h_out + padding - p * dilation;
      if (h_unscaled % stride != 0) continue;
      
      int h_in = h_unscaled / stride;
      if (h_in < 0 || h_in >= in_height) continue;
      
      for (int q = 0; q < kernel_size; ++q) {
        int w_unscaled = w_out + padding - q * dilation;
        if (w_unscaled % stride != 0) continue;
        
        int w_in = w_unscaled / stride;
        if (w_in < 0 || w_in >= in_width) continue;
        
        int input_idx = ((b * in_channels + c) * in_height + h_in) * in_width + w_in;
        int weight_idx = ((c * out_channels + o) * kernel_size + p) * kernel_size + q;
        sum = __fmaf_rn(input[input_idx], weight[weight_idx], sum);
      }
    }
  }
  return sum;
}

__global__ void conv_transpose2d_forward_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

  int w_out = blockIdx.x * blockDim.x + threadIdx.x;
  int h_out = blockIdx.y * blockDim.y + threadIdx.y;
  int batch_channel_idx = blockIdx.z;
  
  int b = batch_channel_idx / out_channels;
  int o = batch_channel_idx % out_channels;

  if (b >= batch_size || o >= out_channels || h_out >= out_height || w_out >= out_width)
    return;

  float out_val = bias[o] + compute_conv_transpose_at_pixel(
    input, weight, b, in_channels, in_height, in_width,
    out_channels, kernel_size, h_out, w_out,
    stride, padding, dilation, o);

  int output_idx = ((b * out_channels + o) * out_height + h_out) * out_width + w_out;
  output[output_idx] = out_val;
}

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Host wrapper to launch the CUDA kernel
void conv_transpose2d_forward(
    at::Tensor input,
    at::Tensor weight,
    at::Tensor bias,
    at::Tensor output,
    int stride,
    int padding,
    int dilation) {
  const auto batch_size = input.size(0);
  const auto in_channels = input.size(1);
  const auto in_height = input.size(2);
  const auto in_width = input.size(3);
  const auto out_channels = weight.size(1);
  const auto kernel_size = weight.size(2);  // assuming square kernel
  const auto out_height = output.size(2);
  const auto out_width = output.size(3);

  dim3 block(16, 16);
  dim3 grid((out_width + block.x - 1) / block.x,
            (out_height + block.y - 1) / block.y,
            batch_size * out_channels);

  conv_transpose2d_forward_kernel<<<grid, block>>>(
      input.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      output.data_ptr<float>(),
      batch_size,
      in_channels,
      out_channels,
      in_height,
      in_width,
      kernel_size,
      out_height,
      out_width,
      stride,
      padding,
      dilation);

  hipDeviceSynchronize();
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("conv_transpose2d_forward", &conv_transpose2d_forward, "Conv Transpose 2D forward (CUDA)");
}
