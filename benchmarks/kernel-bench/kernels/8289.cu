#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/util/Optional.h>

namespace py = pybind11;

// -----------------------------------------------------
// Optimized 1D convolution CUDA kernel with minimized warp divergence
// -----------------------------------------------------
__global__ void conv1d_forward_kernel_optimized(
    const float* __restrict__ x,
    const float* __restrict__ w,
    const float* __restrict__ bias_ptr, // can be null if no bias
    float* __restrict__ y,
    const int N,         // batch size
    const int C_in,      // input channels
    const int L_in,      // input length
    const int C_out,     // output channels
    const int K,         // kernel size
    const int stride,
    const int padding,
    const int dilation,
    const int groups,
    const int L_out      // output length
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = N * C_out * L_out;
    if (idx >= total) return;

    int out_pos = idx % L_out;
    int out_ch  = (idx / L_out) % C_out;
    int n       = idx / (L_out * C_out);

    int group_size_out = C_out / groups;  
    int group_size_in  = C_in  / groups;  
    int group_idx      = out_ch / group_size_out;

    float val = 0.0f;
    for (int local_in_ch = 0; local_in_ch < group_size_in; local_in_ch++) {
        int in_ch = group_idx * group_size_in + local_in_ch;
        for (int k = 0; k < K; k++) {
            int in_pos = out_pos * stride + k * dilation - padding;
            float x_val = (in_pos >= 0 && in_pos < L_in) ? x[n * (C_in * L_in) + in_ch * L_in + in_pos] : 0.0f;
            float w_val = w[out_ch * (group_size_in * K) + local_in_ch * K + k];
            val += x_val * w_val;
        }
    }

    if (bias_ptr) {
        val += bias_ptr[out_ch];
    }

    y[n * (C_out * L_out) + out_ch * L_out + out_pos] = val;
}

// -------------------------------------------------------
// Implementation of conv1d forward with minimized warp divergence
// -------------------------------------------------------
at::Tensor conv1d_forward_impl_optimized(
    const at::Tensor& x,
    const at::Tensor& weight,
    c10::optional<at::Tensor> bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups
)
{
    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.scalar_type() == at::kFloat, "x must be float32");
    TORCH_CHECK(weight.scalar_type() == at::kFloat, "weight must be float32");

    auto x_sizes = x.sizes();
    int64_t N    = x_sizes[0];
    int64_t C_in = x_sizes[1];
    int64_t L_in = x_sizes[2];

    auto w_sizes = weight.sizes();
    int64_t C_out = w_sizes[0];
    int64_t K     = w_sizes[2];

    int64_t L_out = (L_in + 2 * padding - dilation * (K - 1) - 1) / stride + 1;
    TORCH_CHECK(L_out > 0, "Calculated output length is non-positive.");

    auto y = torch::empty({N, C_out, L_out}, x.options().dtype(at::kFloat));

    const float* bias_ptr = nullptr;
    if (bias_opt.has_value() && bias_opt.value().defined()) {
        TORCH_CHECK(bias_opt.value().is_cuda(), "bias must be a CUDA tensor if provided");
        TORCH_CHECK(bias_opt.value().scalar_type() == at::kFloat, "bias must be float32");
        bias_ptr = bias_opt.value().data_ptr<float>();
    }

    int total_threads = N * C_out * L_out;
    int blockSize = 256;
    int gridSize  = (total_threads + blockSize - 1) / blockSize;

    conv1d_forward_kernel_optimized<<<gridSize, blockSize>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        y.data_ptr<float>(),
        N, (int)C_in, (int)L_in, (int)C_out, (int)K,
        (int)stride, (int)padding, (int)dilation, (int)groups, 
        (int)L_out
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "conv1d_forward_kernel_optimized failed: ", hipGetErrorString(err));

    return y;
}

// -----------------------------------------------------
// Pybind11 binding with minimized warp divergence
// -----------------------------------------------------
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        [](at::Tensor x,
           at::Tensor weight,
           py::object bias_obj,
           int64_t stride,
           int64_t padding,
           int64_t dilation,
           int64_t groups) {
            c10::optional<at::Tensor> bias;
            if (!bias_obj.is_none()) {
                bias = bias_obj.cast<at::Tensor>();
            }
            return conv1d_forward_impl_optimized(x, weight, bias, stride, padding, dilation, groups);
        },
        "Optimized 1D Convolution forward (CUDA) with minimized warp divergence"
    );
}