#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>
#include <cmath>

// Kernel optimized using warp-level primitives and shared memory for efficient reduction

template <typename scalar_t, int BLOCK_SIZE>
__global__ void warp_optimized_logsoftmax_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int dim_size) {

    int batch_idx = blockIdx.x;
    const scalar_t* input_row = input + batch_idx * dim_size;
    scalar_t* output_row = output + batch_idx * dim_size;

    __shared__ scalar_t shared_max;
    __shared__ scalar_t shared_sum;
    
    scalar_t local_max = -std::numeric_limits<scalar_t>::infinity();
    scalar_t local_sum = 0;
    
    // Phase 1: Compute maximum value using warp-level reduction
    for (int idx = threadIdx.x; idx < dim_size; idx += BLOCK_SIZE) {
        scalar_t val = input_row[idx];
        local_max = max(local_max, val);
    }

    // Perform warp-level reduction to find the maximum
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        local_max = fmaxf(local_max, __shfl_down_sync(0xFFFFFFFF, local_max, offset));
    }
    
    // Write the warp-level max to shared memory
    if (threadIdx.x % warpSize == 0) {
        atomicMax(&shared_max, local_max);
    }
    __syncthreads();
    max_val = shared_max;

    // Phase 2: Compute sum of exponentials using warp-level reduction
    for (int idx = threadIdx.x; idx < dim_size; idx += BLOCK_SIZE) {
        scalar_t exp_val = exp(input_row[idx] - max_val);
        local_sum += exp_val;
    }

    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        local_sum += __shfl_down_sync(0xFFFFFFFF, local_sum, offset);
    }

    if (threadIdx.x % warpSize == 0) {
        atomicAdd(&shared_sum, local_sum);
    }
    __syncthreads();
    scalar_t sum_val = shared_sum;
    scalar_t log_sum = log(sum_val);

    // Phase 3: Compute final LogSoftmax values
    for (int idx = threadIdx.x; idx < dim_size; idx += BLOCK_SIZE) {
        output_row[idx] = (input_row[idx] - max_val) - log_sum;
    }
}

// Host function

torch::Tensor warp_optimized_logsoftmax_cuda_forward(torch::Tensor input, int64_t dim) {
    TORCH_CHECK(input.is_cuda(), "input must be a CUDA tensor");
    TORCH_CHECK(
        input.scalar_type() == torch::kFloat32 || input.scalar_type() == torch::kFloat64,
        "input must be float32 or float64");

    int64_t ndim = input.dim();
    TORCH_CHECK(dim >= -ndim && dim < ndim, "dim out of range");
    dim = dim >= 0 ? dim : dim + ndim;

    std::vector<int64_t> permute_dims;
    for (int64_t i = 0; i < ndim; ++i) {
        if (i != dim) {
            permute_dims.push_back(i);
        }
    }
    permute_dims.push_back(dim);

    input = input.permute(permute_dims).contiguous();
    int64_t batch_size = input.numel() / input.size(-1);
    int64_t dim_size = input.size(-1);

    auto output = torch::empty_like(input);

    int optimal_block_size = 256;
    if (dim_size <= 32) {
        optimal_block_size = 32;
    } else if (dim_size <= 64) {
        optimal_block_size = 64;
    } else if (dim_size <= 128) {
        optimal_block_size = 128;
    } else if (dim_size <= 256) {
        optimal_block_size = 256;
    } else if (dim_size <= 512) {
        optimal_block_size = 512;
    } else {
        optimal_block_size = 512;
    }

    const int blocks = batch_size;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "warp_optimized_logsoftmax_cuda_forward", ([&] {
        if (optimal_block_size == 32) {
            warp_optimized_logsoftmax_kernel<scalar_t, 32><<<blocks, 32>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else if (optimal_block_size == 64) {
            warp_optimized_logsoftmax_kernel<scalar_t, 64><<<blocks, 64>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else if (optimal_block_size == 128) {
            warp_optimized_logsoftmax_kernel<scalar_t, 128><<<blocks, 128>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else if (optimal_block_size == 256) {
            warp_optimized_logsoftmax_kernel<scalar_t, 256><<<blocks, 256>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else if (optimal_block_size == 512) {
            warp_optimized_logsoftmax_kernel<scalar_t, 512><<<blocks, 512>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        }
    }));

    std::vector<int64_t> inverse_permute_dims(ndim);
    for (size_t i = 0; i < permute_dims.size(); ++i) {
        inverse_permute_dims[permute_dims[i]] = i;
    }
    output = output.permute(inverse_permute_dims);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &warp_optimized_logsoftmax_cuda_forward, "Warp Optimized LogSoftmax forward (CUDA)");
}
