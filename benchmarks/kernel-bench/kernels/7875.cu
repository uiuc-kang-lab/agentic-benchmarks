#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Declare constant memory for weights (limited to 64KB on most CUDA devices)
__constant__ float const_weight[16384]; // 64KB / 4 bytes = 16384 float elements

__global__ void conv2d_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int out_channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int stride,
    const int padding) {

    const int out_h = (height + 2 * padding - kernel_h) / stride + 1;
    const int out_w = (width + 2 * padding - kernel_w) / stride + 1;
    const int total = batch_size * out_channels * out_h * out_w;
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index >= total) return;

    const int b = index / (out_channels * out_h * out_w);
    const int rem = index % (out_channels * out_h * out_w);
    const int oc = rem / (out_h * out_w);
    const int rem2 = rem % (out_h * out_w);
    const int h = rem2 / out_w;
    const int w = rem2 % out_w;

    float sum = 0.0f;

    const int h_start = max(0, h * stride - padding);
    const int h_end = min(kernel_h, height - h * stride + padding);
    const int w_start = max(0, -w * stride + padding);
    const int w_end = min(kernel_w, width - w * stride + padding);

    #pragma unroll 4
    for (int ic = 0; ic < in_channels; ++ic) {
        for (int kh = h_start; kh < h_end; ++kh) {
            const int h_in = h * stride + kh - padding;
            #pragma unroll 4
            for (int kw = w_start; kw < w_end; ++kw) {
                const int w_in = w * stride + kw - padding;
                const float input_val = __ldg(&input[((b * in_channels + ic) * height + h_in) * width + w_in]);
                const int weight_idx = ((oc * in_channels + ic) * kernel_h + kh) * kernel_w + kw;
                sum += input_val * const_weight[weight_idx];
            }
        }
    }

    output[((b * out_channels + oc) * out_h + h) * out_w + w] = sum;
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    TORCH_CHECK(x.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "Weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "Input must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "Weight must be contiguous");
    
    if (dilation != 1 || groups != 1) {
        return torch::conv2d(x, weight, bias,
                           {stride, stride}, {padding, padding}, {dilation, dilation}, groups);
    }

    auto batch_size = x.size(0);
    auto in_channels = x.size(1);
    auto height = x.size(2);
    auto width = x.size(3);
    auto out_channels = weight.size(0);
    auto kernel_h = weight.size(2);
    auto kernel_w = weight.size(3);
    
    // Check if weights fit in constant memory
    TORCH_CHECK(weight.numel() <= 16384, "Weight tensor too large for constant memory");
    
    // Copy weights to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(const_weight), weight.data_ptr<float>(), 
                       weight.numel() * sizeof(float));
    
    const int out_h = (height + 2 * padding - kernel_h) / stride + 1;
    const int out_w = (width + 2 * padding - kernel_w) / stride + 1;
    
    auto output = torch::zeros({batch_size, out_channels, out_h, out_w}, x.options());

    constexpr int BLOCK_SIZE = 256;
    const int total = batch_size * out_channels * out_h * out_w;
    const int blocks = (total + BLOCK_SIZE - 1) / BLOCK_SIZE;

    conv2d_kernel<<<blocks, BLOCK_SIZE>>>(
        x.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        out_channels,
        height,
        width,
        kernel_h,
        kernel_w,
        stride,
        padding
    );

    if (bias.has_value()) {
        output.add_(bias.value().view({1, -1, 1, 1}));
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA 2D Convolution");
}