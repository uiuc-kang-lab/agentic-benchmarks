#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 64  // Experimenting with a larger block size

// CUDA kernel for triangular matrix multiplication with optimized block size
__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                       const float* __restrict__ B,
                                       float* __restrict__ C,
                                       const int N) {
    __shared__ float shA[TILE_SIZE][TILE_SIZE];
    __shared__ float shB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    if (row >= N || col >= N) return;
    // For lower triangular matrices, elements where row < col are zero
    if (row < col) {
        C[row * N + col] = 0.0f;
        return;
    }

    float sum = 0.0f;

    // Determine tile range relevant for computation
    int t_start = col / TILE_SIZE;
    int t_end   = row / TILE_SIZE;

    // Loop over tiles contributing to the result
    #pragma unroll
    for (int t = t_start; t <= t_end; t++) {
        // Load A tile: only load if the column index is within bounds and valid for lower triangular
        int a_col = t * TILE_SIZE + threadIdx.x;
        if (a_col < N && a_col <= row)
            shA[threadIdx.y][threadIdx.x] = A[row * N + a_col];
        else
            shA[threadIdx.y][threadIdx.x] = 0.0f;

        // Load B tile: only load if the row index is within bounds and meets triangular condition
        int b_row = t * TILE_SIZE + threadIdx.y;
        if (b_row < N && b_row >= col)
            shB[threadIdx.y][threadIdx.x] = B[b_row * N + col];
        else
            shB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Determine the effective k range in this tile
        int k_begin = t * TILE_SIZE;
        if (k_begin < col) k_begin = col;
        int k_end = (t + 1) * TILE_SIZE;
        if (k_end > row + 1) k_end = row + 1;
        int iter = k_end - k_begin;

        // If the entire tile is available, unroll fully for maximum performance
        if (iter == TILE_SIZE) {
            #pragma unroll
            for (int i = 0; i < TILE_SIZE; i++) {
                sum += shA[threadIdx.y][i] * shB[i][threadIdx.x];
            }
        } else {
            #pragma unroll
            for (int k = k_begin; k < k_end; k++) {
                int local_k = k - t * TILE_SIZE;
                sum += shA[threadIdx.y][local_k] * shB[local_k][threadIdx.x];
            }
        }
        __syncthreads();
    }

    C[row * N + col] = sum;
}

// C++ interface exposed to PyTorch
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "A and B must be CUDA tensors");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D tensors");
    TORCH_CHECK(A.size(0) == A.size(1) && B.size(0) == B.size(1), "A and B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    const int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    triangular_mm_kernel<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA) with optimized block size");
}
