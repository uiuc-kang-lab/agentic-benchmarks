#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel: optimized for memory coalescing
__global__ void diag_matmul_kernel_coalesced(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    const int64_t N,
    const int64_t M
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N) {
        float a_val = A[row];
        for (int col = 0; col < M; ++col) {
            int idx = row * M + col;
            C[idx] = a_val * B[idx];
        }
    }
}

// Forward function that wraps our CUDA kernel
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.dim() == 1, "A must be a 1D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == B.size(0),
                "Dimension mismatch: A.size(0) must match B.size(0)");

    // Ensure inputs are on contiguous memory
    A = A.contiguous();
    B = B.contiguous();

    int64_t N = A.size(0);
    int64_t M = B.size(1);

    // Create an output tensor with the same device and type as B
    auto C = torch::empty({N, M}, B.options());

    // Configure and launch the kernel
    const int64_t threads = 256;
    const int64_t blocks = (N + threads - 1) / threads;
    diag_matmul_kernel_coalesced<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N,
        M
    );

    return C;
}

// Create the PyTorch extension module
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Diagonal matrix multiplication of A and B on the GPU");
}