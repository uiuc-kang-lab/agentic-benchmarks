#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/util/Optional.h>

namespace py = pybind11;

__global__ void conv1d_forward_kernel(
    const float* __restrict__ x,
    const float* __restrict__ w,
    const float* __restrict__ bias_ptr,
    float* __restrict__ y,
    const int N,
    const int C_in,
    const int L_in,
    const int C_out,
    const int K,
    const int stride,
    const int padding,
    const int dilation,
    const int groups,
    const int L_out,
    const int valid_start,
    const int valid_end
) {
    const int out_ch = blockIdx.x;
    const int out_pos = blockIdx.y * blockDim.x + threadIdx.x;
    const int n = blockIdx.z;
    
    if (out_pos >= L_out) return;

    const int group_size_out = C_out / groups;
    const int group_size_in = C_in / groups;
    const int group_idx = out_ch / group_size_out;
    const int group_start = group_idx * group_size_in;
    
    const int batch_offset = n * (C_in * L_in);
    const int weight_offset = out_ch * (group_size_in * K);
    const int output_offset = n * (C_out * L_out) + out_ch * L_out + out_pos;
    
    const int in_pos_base = out_pos * stride - padding;
    float sum = 0.0f;

    #pragma unroll 4
    for (int local_in_ch = 0; local_in_ch < group_size_in; ++local_in_ch) {
        const int in_ch = group_start + local_in_ch;
        const int in_offset = batch_offset + in_ch * L_in;
        const int w_offset = weight_offset + local_in_ch * K;
        
        #pragma unroll
        for (int k = 0; k < K; ++k) {
            const int in_pos = in_pos_base + k * dilation;
            const bool valid_pos = (in_pos >= 0) && (in_pos < L_in);
            const float x_val = valid_pos ? __ldg(&x[in_offset + in_pos]) : 0.0f;
            const float w_val = __ldg(&w[w_offset + k]);
            sum = fmaf(x_val, w_val, sum);
        }
    }

    sum += (bias_ptr != nullptr) ? __ldg(&bias_ptr[out_ch]) : 0.0f;
    y[output_offset] = sum;
}

at::Tensor conv1d_forward_impl(
    const at::Tensor& x,
    const at::Tensor& weight,
    c10::optional<at::Tensor> bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups
) {
    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.scalar_type() == at::kFloat, "x must be float32");
    TORCH_CHECK(weight.scalar_type() == at::kFloat, "weight must be float32");

    auto x_sizes = x.sizes();
    int64_t N    = x_sizes[0];
    int64_t C_in = x_sizes[1];
    int64_t L_in = x_sizes[2];

    auto w_sizes = weight.sizes();
    int64_t C_out = w_sizes[0];
    int64_t K     = w_sizes[2];

    int64_t L_out = (L_in + 2 * padding - dilation * (K - 1) - 1) / stride + 1;
    TORCH_CHECK(L_out > 0, "Calculated output length is non-positive.");

    auto y = torch::empty({N, C_out, L_out}, x.options().dtype(at::kFloat));

    const float* bias_ptr = nullptr;
    if (bias_opt.has_value() && bias_opt.value().defined()) {
        TORCH_CHECK(bias_opt.value().is_cuda(), "bias must be a CUDA tensor if provided");
        TORCH_CHECK(bias_opt.value().scalar_type() == at::kFloat, "bias must be float32");
        bias_ptr = bias_opt.value().data_ptr<float>();
    }

    const int valid_start = padding;
    const int valid_end = L_in + padding;

    dim3 blockSize(256);
    dim3 gridSize(C_out, (L_out + blockSize.x - 1) / blockSize.x, N);

    conv1d_forward_kernel<<<gridSize, blockSize>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        y.data_ptr<float>(),
        N, (int)C_in, (int)L_in, (int)C_out, (int)K,
        (int)stride, (int)padding, (int)dilation, (int)groups,
        (int)L_out, valid_start, valid_end
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "conv1d_forward_kernel failed: ", hipGetErrorString(err));

    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        [](at::Tensor x,
           at::Tensor weight,
           py::object bias_obj,
           int64_t stride,
           int64_t padding,
           int64_t dilation,
           int64_t groups) {
            c10::optional<at::Tensor> bias;
            if (!bias_obj.is_none()) {
                bias = bias_obj.cast<at::Tensor>();
            }
            return conv1d_forward_impl(x, weight, bias, stride, padding, dilation, groups);
        },
        "Divergence-free 1D Convolution forward (CUDA)"
    );
}