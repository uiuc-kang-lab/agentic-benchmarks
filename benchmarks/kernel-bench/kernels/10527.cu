#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void cumsum_kernel_coalesced(const float* __restrict__ input, float* __restrict__ output,
                                         int outer_size, int inner_size, int stride) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= outer_size * stride) return;
    int inner_idx = tid % stride;
    int outer_idx = tid / stride;

    int idx = outer_idx * stride * inner_size + inner_idx * inner_size;
    float sum = 0.0f;
    for (int i = 0; i < inner_size; ++i) {
        float val = __ldg(&input[idx + i]);
        sum += val;
        output[idx + i] = sum;
    }
}

__global__ void cumsum_kernel_unroll_align(const float* __restrict__ input, float* __restrict__ output,
                                            int dim, int outer_size, int inner_size, int stride) {
    int inner_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (inner_idx >= inner_size) return;

    // Use shared memory for partial sums to reduce register pressure
    __shared__ float partial_sums[256];  // Assuming blockSize <= 256
    
    for (int outer_idx = 0; outer_idx < outer_size; ++outer_idx) {
        int base_idx = outer_idx * stride * inner_size + inner_idx;
        float sum = 0.0f;
        
        // Process in chunks to reduce register pressure
        #pragma unroll 4
        for (int i = 0; i < stride; i += 4) {
            // Load 4 elements at a time
            float val1 = (i < stride) ? __ldg(&input[base_idx + i * inner_size]) : 0.0f;
            float val2 = (i + 1 < stride) ? __ldg(&input[base_idx + (i + 1) * inner_size]) : 0.0f;
            float val3 = (i + 2 < stride) ? __ldg(&input[base_idx + (i + 2) * inner_size]) : 0.0f;
            float val4 = (i + 3 < stride) ? __ldg(&input[base_idx + (i + 3) * inner_size]) : 0.0f;
            
            // Update partial sums
            sum += val1;
            output[base_idx + i * inner_size] = sum;
            if (i + 1 < stride) {
                sum += val2;
                output[base_idx + (i + 1) * inner_size] = sum;
            }
            if (i + 2 < stride) {
                sum += val3;
                output[base_idx + (i + 2) * inner_size] = sum;
            }
            if (i + 3 < stride) {
                sum += val4;
                output[base_idx + (i + 3) * inner_size] = sum;
            }
        }
    }
}

torch::Tensor forward(torch::Tensor x, int dim) {
    CHECK_INPUT(x);

    auto output = torch::empty_like(x);
    int ndim = x.dim();
    dim = (dim + ndim) % ndim;

    int outer_size = 1;
    for (int i = 0; i < dim; ++i) {
        outer_size *= x.size(i);
    }

    int inner_size = 1;
    for (int i = dim + 1; i < ndim; ++i) {
        inner_size *= x.size(i);
    }

    int stride = x.size(dim);
    int blockSize = 256;
    int numBlocks = (outer_size * stride + blockSize - 1) / blockSize;

    cumsum_kernel_unroll_align<<<numBlocks, blockSize>>>(
        x.data_ptr<float>(), output.data_ptr<float>(), dim, outer_size, inner_size, stride
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA cumulative sum with memory coalescing and unrolling");
}