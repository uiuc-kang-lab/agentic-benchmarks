#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

// Kernel using shared memory tiling for predictions and targets
// This kernel loads contiguous tiles of data into shared memory, reducing global memory latency
// and then computes the squared differences. Reduction is performed in shared memory before
// atomic accumulation into the global sum.

template <typename scalar_t>
__global__ void mse_shared_forward_kernel(
    const scalar_t* __restrict__ preds,
    const scalar_t* __restrict__ tgts,
    double* __restrict__ sum_out,
    const int64_t num_elements
) {
    // Shared memory tiles for predictions and targets
    __shared__ scalar_t pred_tile[BLOCK_SIZE];
    __shared__ scalar_t tgt_tile[BLOCK_SIZE];

    double local_sum = 0.0;
    int tid = threadIdx.x;
    
    // Each block processes multiple contiguous tiles
    // Compute starting offset for this block's tiles
    for (int base = blockIdx.x * blockDim.x; base < num_elements; base += blockDim.x * gridDim.x) {
        int index = base + tid;
        // Load data into shared memory if within bounds
        if (index < num_elements) {
            pred_tile[tid] = preds[index];
            tgt_tile[tid] = tgts[index];
        }
        __syncthreads();  // ensure data is loaded before usage
        
        // Each thread computes its contribution using the tile data
        if (index < num_elements) {
            double diff = static_cast<double>(pred_tile[tid]) - static_cast<double>(tgt_tile[tid]);
            local_sum += diff * diff;
        }
        __syncthreads();  // ensure all threads have finished using shared memory
    }

    // Reduction within the block using shared memory for accumulation
    __shared__ double sdata[BLOCK_SIZE];
    sdata[tid] = local_sum;
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // Atomic add the block's result to the global accumulator
    if (tid == 0) {
        atomicAdd(sum_out, sdata[0]);
    }
}

// Host function that dispatches the CUDA kernel
torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.is_cuda(), "predictions must be a CUDA tensor");
    TORCH_CHECK(targets.is_cuda(), "targets must be a CUDA tensor");
    TORCH_CHECK(predictions.numel() == targets.numel(),
                "predictions and targets must have the same number of elements");

    const int64_t num_elements = predictions.numel();
    // Use double precision for accumulation to ensure correctness
    auto accumulator = torch::zeros({1}, predictions.options().dtype(at::kDouble));

    const int grid_size = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    AT_DISPATCH_FLOATING_TYPES(predictions.scalar_type(), "mse_shared_forward_cuda", [&] {
        mse_shared_forward_kernel<scalar_t><<<grid_size, BLOCK_SIZE>>>(
            predictions.data_ptr<scalar_t>(),
            targets.data_ptr<scalar_t>(),
            accumulator.data_ptr<double>(),
            num_elements
        );
    });

    // Compute final mean by dividing the accumulated squared error by the number of elements
    auto result = accumulator.div_(static_cast<double>(num_elements));
    return result.to(predictions.dtype());
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Mean Squared Error (MSE) forward (CUDA) using shared memory tiling");
}
