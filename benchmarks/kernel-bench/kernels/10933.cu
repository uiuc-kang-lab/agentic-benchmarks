#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static const int BLOCK_SIZE = 256;

// CUDA kernel using a grid-stride loop with correct boundary handling
template <typename scalar_t>
__global__ void stride_mse_kernel(
    const scalar_t* __restrict__ preds,
    const scalar_t* __restrict__ tgts,
    double* __restrict__ sum_out,
    const int64_t num_elements
) {
    // Shared memory for reduction
    __shared__ double shmem[BLOCK_SIZE];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    double local_sum = 0.0;

    // Grid-stride loop to handle workloads larger than available threads
    for (; idx < num_elements; idx += stride) {
        // Verify boundary before reading
        double diff = static_cast<double>(preds[idx]) - static_cast<double>(tgts[idx]);
        local_sum += diff * diff;
    }

    // Store local sum to shared memory
    shmem[threadIdx.x] = local_sum;
    __syncthreads();

    // Intra-block reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shmem[threadIdx.x] += shmem[threadIdx.x + s];
        }
        __syncthreads();
    }

    // The first thread of each block updates the global accumulator
    if (threadIdx.x == 0) {
        atomicAdd(sum_out, shmem[0]);
    }
}

// Host function to launch the kernel

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.is_cuda(), "predictions must be a CUDA tensor");
    TORCH_CHECK(targets.is_cuda(), "targets must be a CUDA tensor");
    TORCH_CHECK(predictions.numel() == targets.numel(), "predictions and targets must have the same number of elements");

    const int64_t num_elements = predictions.numel();
    auto accumulator = torch::zeros({1}, predictions.options().dtype(at::kDouble));

    // Determine grid size ensuring we don't oversubscribe
    int grid_size = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;
    grid_size = (grid_size < 1024) ? grid_size : 1024;

    AT_DISPATCH_FLOATING_TYPES(predictions.scalar_type(), "stride_mse_cuda", ([&] {
        stride_mse_kernel<scalar_t><<<grid_size, BLOCK_SIZE>>>(
            predictions.data_ptr<scalar_t>(),
            targets.data_ptr<scalar_t>(),
            accumulator.data_ptr<double>(),
            num_elements);
    }));

    // Compute the mean squared error
    auto result = accumulator.div_(static_cast<double>(num_elements));
    return result.to(predictions.dtype());
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Stride Loop Reduction MSE Forward (CUDA)");
}
