#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

__global__ void einsum_kernel_stride(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int BATCH, int I, int J, int L, int K
) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int total_elements = BATCH * I * J * K;

    for (int idx = global_idx; idx < total_elements; idx += stride) {
        int k = idx % K;
        int remainder = idx / K;
        int j = remainder % J;
        remainder /= J;
        int i = remainder % I;
        int b = remainder / I;

        float sum = 0.0f;
        for(int l = 0; l < L; ++l) {
            int a_offset = b * I*J*L + i*J*L + j*L + l;
            int b_offset = l*K + k;
            sum += A[a_offset] * B[b_offset];
        }
        
        C[idx] = sum;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(A.dim() == 4, "A must be 4D");
    TORCH_CHECK(B.dim() == 2, "B must be 2D");
    TORCH_CHECK(A.size(3) == B.size(0), "Dimension mismatch in l");

    int BATCH = A.size(0), I = A.size(1), J = A.size(2), L = A.size(3);
    int K = B.size(1);
    
    auto C = torch::zeros({BATCH, I, J, K}, A.options());
    int total_elements = BATCH * I * J * K;
    
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;
    
    einsum_kernel_stride<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        BATCH, I, J, L, K
    );
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "4D tensor-matrix multiplication with stride loop (CUDA)");
}