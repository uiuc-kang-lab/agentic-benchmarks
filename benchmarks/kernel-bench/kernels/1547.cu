#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32

__global__ void matmul_kernel(const float* __restrict__ A,
                             const float* __restrict__ B,
                             float* __restrict__ C,
                             int N) {
    __shared__ float s_A[TILE_SIZE][TILE_SIZE];
    __shared__ float s_B[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // Stride loop for output matrix coverage
    for (int row = blockIdx.y * blockDim.y + ty; row < N; row += blockDim.y * gridDim.y) {
        for (int col = blockIdx.x * blockDim.x + tx; col < N; col += blockDim.x * gridDim.x) {
            float value = 0;

            for (int i = 0; i < (N + TILE_SIZE - 1) / TILE_SIZE; ++i) {
                // Prefetch next tiles' indices
                int next_a_col = (i + 1) * TILE_SIZE + tx;
                int next_b_row = (i + 1) * TILE_SIZE + ty;

                // Load current tiles
                int a_col = i * TILE_SIZE + tx;
                int b_row = i * TILE_SIZE + ty;
                s_A[ty][tx] = (row < N && a_col < N) ? A[row * N + a_col] : 0.0f;
                s_B[ty][tx] = (b_row < N && col < N) ? B[b_row * N + col] : 0.0f;

                __syncthreads();

                // Start computing current tile while prefetching next tiles
                float next_a = (i < ((N + TILE_SIZE - 1) / TILE_SIZE) - 1) ? 
                    ((row < N && next_a_col < N) ? A[row * N + next_a_col] : 0.0f) : 0.0f;
                float next_b = (i < ((N + TILE_SIZE - 1) / TILE_SIZE) - 1) ? 
                    ((next_b_row < N && col < N) ? B[next_b_row * N + col] : 0.0f) : 0.0f;

                // Compute on current tiles
                #pragma unroll
                for (int k = 0; k < TILE_SIZE; ++k)
                    value += s_A[ty][k] * s_B[k][tx];

                __syncthreads();

                // Store prefetched data for next iteration
                if (i < ((N + TILE_SIZE - 1) / TILE_SIZE) - 1) {
                    s_A[ty][tx] = next_a;
                    s_B[ty][tx] = next_b;
                }
            }

            // Write result with boundary check
            if (row < N && col < N)
                C[row * N + col] = value;
        }
    }
}

void matmul_with_streams(torch::Tensor A, torch::Tensor B, torch::Tensor C, hipStream_t stream) {
    int N = A.size(0);

    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    
    // Reduce grid dimensions to force stride loops
    grid.x = min(grid.x, 65535);
    grid.y = min(grid.y, 65535);

    matmul_kernel<<<grid, block, 0, stream>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), N);
}

// C++ interface
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(A.size(0) == A.size(1) && B.size(0) == B.size(1), "Matrices must be square");

    int N = A.size(0);
    auto C = torch::zeros({N, N}, A.options());

    // Create a CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Launch the kernel with the stream
    matmul_with_streams(A, B, C, stream);

    // Synchronize the stream to ensure completion
    hipStreamSynchronize(stream);

    // Destroy the stream
    hipStreamDestroy(stream);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix Multiplication with Stream Overlap (CUDA)");
}