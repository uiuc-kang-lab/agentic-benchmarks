#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32

namespace {
__device__ inline bool thread_in_bounds(int row, int col, int N) {
    return (row < N && col < N);
}

__device__ inline float compute_element(
    const float* __restrict__ A,
    const float* __restrict__ B,
    int row,
    int col,
    int N
) {
    float sum = 0.0f;
    #pragma unroll
    for(int k = col; k <= row; ++k) {
        sum += __ldg(&A[row * N + k]) * __ldg(&B[k * N + col]);
    }
    return sum;
}
} // anonymous namespace

__global__ void uniform_control_flow_triangular_mm_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int N
) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int row = static_cast<int>(blockIdx.y) * TILE_SIZE + static_cast<int>(threadIdx.y);
    int col = static_cast<int>(blockIdx.x) * TILE_SIZE + static_cast<int>(threadIdx.x);

    float sum = 0.0f;

    for (int t = static_cast<int>(blockIdx.x); t <= static_cast<int>(blockIdx.y); ++t) {
        int tRow = static_cast<int>(blockIdx.y) * TILE_SIZE + static_cast<int>(threadIdx.y);
        int tCol = t * TILE_SIZE + static_cast<int>(threadIdx.x);
        As[threadIdx.y][threadIdx.x] = (thread_in_bounds(tRow, tCol, N) && row >= tCol) ? __ldg(&A[tRow * N + tCol]) : 0.0f;

        tRow = t * TILE_SIZE + static_cast<int>(threadIdx.y);
        tCol = static_cast<int>(blockIdx.x) * TILE_SIZE + static_cast<int>(threadIdx.x);
        Bs[threadIdx.y][threadIdx.x] = (thread_in_bounds(tRow, tCol, N) && tRow >= col) ? __ldg(&B[tRow * N + tCol]) : 0.0f;

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (thread_in_bounds(row, col, N)) {
        C[row * N + col] = (row >= col) ? sum : 0.0f;
    }
}

at::Tensor forward(const at::Tensor& A, const at::Tensor& B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(A.size(0) == A.size(1) && B.size(0) == B.size(1), "Matrices must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "Matrices must be same size");

    int N = static_cast<int>(A.size(0));
    auto C = torch::empty_like(A);

    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);
    dim3 block(TILE_SIZE, TILE_SIZE);

    uniform_control_flow_triangular_mm_kernel<<<grid, block>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), N);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA error: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Uniform control flow triangular matrix multiplication (CUDA)");
}