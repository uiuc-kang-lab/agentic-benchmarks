#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Adjusted kernel parameters for improved performance based on hardware requirements

template <typename scalar_t>
__global__ void triplet_margin_loss_kernel(
    const scalar_t* anchor,
    const scalar_t* positive,
    const scalar_t* negative,
    scalar_t* output,
    const float margin,
    const int batch_size,
    const int feat_size) {
    
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int batch_idx = tid / feat_size;
    const int feat_idx = tid % feat_size;
    const int warp_size = 32;
    const unsigned mask = 0xffffffff;
    
    if (batch_idx < batch_size && feat_idx < feat_size) {
        const int idx = batch_idx * feat_size + feat_idx;
        const scalar_t a = anchor[idx];
        const scalar_t p = positive[idx];
        const scalar_t n = negative[idx];
        
        // Compute distance components
        const scalar_t d_pos = a - p;
        const scalar_t d_neg = a - n;
        
        // Squared distances
        scalar_t dist_pos = d_pos * d_pos;
        scalar_t dist_neg = d_neg * d_neg;
        
        // Use shared memory for reduction
        __shared__ scalar_t shared_pos[512];
        __shared__ scalar_t shared_neg[512];
        
        // Initialize shared memory
        shared_pos[threadIdx.x] = dist_pos;
        shared_neg[threadIdx.x] = dist_neg;
        __syncthreads();
        
        // Parallel reduction in shared memory
        for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                shared_pos[threadIdx.x] += shared_pos[threadIdx.x + stride];
                shared_neg[threadIdx.x] += shared_neg[threadIdx.x + stride];
            }
            __syncthreads();
        }
        
        if (feat_idx == 0) {
            // Final loss computation for this batch element
            const scalar_t loss = max(scalar_t(0.0), sqrt(dist_pos) - sqrt(dist_neg) + margin);
            output[batch_idx] = loss;
        }
    }
}

torch::Tensor triplet_margin_loss_cuda(
    torch::Tensor anchor,
    torch::Tensor positive,
    torch::Tensor negative,
    float margin) {
    
    TORCH_CHECK(anchor.device().is_cuda(), "anchor must be a CUDA tensor");
    TORCH_CHECK(positive.device().is_cuda(), "positive must be a CUDA tensor");
    TORCH_CHECK(negative.device().is_cuda(), "negative must be a CUDA tensor");
    
    const int batch_size = anchor.size(0);
    const int feat_size = anchor.size(1);
    auto output = torch::zeros({batch_size}, anchor.options());
    
    const int threads = 512;  // optimized block size
    const int blocks = (batch_size * feat_size + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES(anchor.scalar_type(), "triplet_margin_loss_kernel", ([&] {
        triplet_margin_loss_kernel<scalar_t><<<blocks, threads>>>(
            anchor.data_ptr<scalar_t>(),
            positive.data_ptr<scalar_t>(),
            negative.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            margin,
            batch_size,
            feat_size);
    }));
    
    return output.mean();
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &triplet_margin_loss_cuda, "Triplet margin loss forward (CUDA)");
}
