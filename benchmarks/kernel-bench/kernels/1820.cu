#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   const int N) {
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int row = blockIdx.y * BLOCK_SIZE + ty;
    const int col = blockIdx.x * BLOCK_SIZE + tx;
    
    float sum = 0.0f;
    
    if (row < N && col < N) {
        if (row < col) {
            C[row * N + col] = 0.0f;
            return;
        }
        
        // Number of tile iterations for this row-col pair
        const int numTiles = (min(row, N-1) - col + BLOCK_SIZE) / BLOCK_SIZE;
        
        for (int t = 0; t < numTiles; t++) {
            const int tileStart = col + t * BLOCK_SIZE;
            
            // Load tile into shared memory
            if (row < N && (tileStart + tx) <= row) {
                As[ty][tx] = A[row * N + (tileStart + tx)];
            } else {
                As[ty][tx] = 0.0f;
            }
            
            if ((tileStart + ty) < N && col < N) {
                Bs[ty][tx] = B[(tileStart + ty) * N + col];
            } else {
                Bs[ty][tx] = 0.0f;
            }
            
            __syncthreads();
            
            // Compute partial sum for this tile
            #pragma unroll 8
            for (int k = 0; k < BLOCK_SIZE; k++) {
                if ((tileStart + k) <= row) {
                    sum += As[ty][k] * Bs[k][tx];
                }
            }
            
            __syncthreads();
        }
        
        if (row < N && col < N && row >= col) {
            C[row * N + col] = sum;
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, 
                   (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}