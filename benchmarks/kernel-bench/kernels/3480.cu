#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Store frequently accessed constants in constant memory
__constant__ float sqrt_2_inv = 0.7071067811865475f;

// Device GELU function specializations
template <typename scalar_t>
__device__ inline scalar_t gelu_function(scalar_t x);

template <>
__device__ inline float gelu_function<float>(float x) {
    return x * 0.5f * (1.0f + erff(x * sqrt_2_inv));
}

template <>
__device__ inline double gelu_function<double>(double x) {
    return x * 0.5 * (1.0 + erf(x * sqrt_2_inv));
}

// Vectorized kernel with constant memory usage
template <typename scalar_t, int VEC_SIZE>
__global__ void gelu_vectorized_kernel(const scalar_t* __restrict__ input,
                                      scalar_t* __restrict__ output,
                                      size_t n) {
    const int tid = blockIdx.x * blockDim.x * VEC_SIZE + threadIdx.x * VEC_SIZE;
    if (tid + VEC_SIZE <= n) {
        // Process full vector without boundary checks
        #pragma unroll
        for (int i = 0; i < VEC_SIZE; ++i) {
            scalar_t val = input[tid + i];
            output[tid + i] = gelu_function(val);
        }
    } else if (tid < n) {
        // Process remaining elements with boundary checks
        for (int i = 0; i < VEC_SIZE; ++i) {
            int index = tid + i;
            if (index < n) {
                scalar_t val = input[index];
                output[index] = gelu_function(val);
            }
        }
    }
}

torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
    auto y = torch::empty_like(x);
    const size_t n = x.numel();
    
    constexpr int VEC_SIZE = 4;
    const int threads = 256;
    const int blocks = (n + threads * VEC_SIZE - 1) / (threads * VEC_SIZE);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "gelu_forward", [&] {
        gelu_vectorized_kernel<scalar_t, VEC_SIZE>
            <<<blocks, threads>>>(x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), n);
    });

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA error: ", hipGetErrorString(err));
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "GELU forward (CUDA)");
}