#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Kernel using uniform control flow to minimize warp divergence

template <typename scalar_t>
__global__ void uniform_control_flow_triplet_kernel(
    const scalar_t* __restrict__ anchor,
    const scalar_t* __restrict__ positive,
    const scalar_t* __restrict__ negative,
    scalar_t* __restrict__ output,
    const float margin,
    const int batch_size,
    const int feat_size) {

    int batch_idx = blockIdx.x;
    if (batch_idx >= batch_size) return;

    int tid = threadIdx.x;
    int base_idx = batch_idx * feat_size;
    scalar_t local_dist_pos = 0;
    scalar_t local_dist_neg = 0;

    // Use vectorized loads for read-only global memory accesses with __ldg()
    if constexpr (std::is_same<scalar_t, float>::value) {
        using vec_t = float4;
        constexpr int vec_size = 4;
        int vectorized_length = feat_size / vec_size;
        int remainder = feat_size % vec_size;
        const vec_t* anchor_vec = reinterpret_cast<const vec_t*>(anchor + base_idx);
        const vec_t* positive_vec = reinterpret_cast<const vec_t*>(positive + base_idx);
        const vec_t* negative_vec = reinterpret_cast<const vec_t*>(negative + base_idx);

        for (int i = tid; i < vectorized_length; i += blockDim.x) {
            vec_t a_vec = __ldg(&anchor_vec[i]);
            vec_t p_vec = __ldg(&positive_vec[i]);
            vec_t n_vec = __ldg(&negative_vec[i]);
            
            float diff0 = a_vec.x - p_vec.x;
            float diff1 = a_vec.y - p_vec.y;
            float diff2 = a_vec.z - p_vec.z;
            float diff3 = a_vec.w - p_vec.w;
            local_dist_pos += diff0 * diff0 + diff1 * diff1 + diff2 * diff2 + diff3 * diff3;
            
            float diff0n = a_vec.x - n_vec.x;
            float diff1n = a_vec.y - n_vec.y;
            float diff2n = a_vec.z - n_vec.z;
            float diff3n = a_vec.w - n_vec.w;
            local_dist_neg += diff0n * diff0n + diff1n * diff1n + diff2n * diff2n + diff3n * diff3n;
        }

        int offset = vectorized_length * vec_size;
        for (int i = tid; i < remainder; i += blockDim.x) {
            int idx = base_idx + offset + i;
            float a = __ldg(&anchor[idx]);
            float p = __ldg(&positive[idx]);
            float n = __ldg(&negative[idx]);
            float diff = a - p;
            local_dist_pos += diff * diff;
            float diffn = a - n;
            local_dist_neg += diffn * diffn;
        }
    } else if constexpr (std::is_same<scalar_t, double>::value) {
        using vec_t = double2;
        constexpr int vec_size = 2;
        int vectorized_length = feat_size / vec_size;
        int remainder = feat_size % vec_size;
        const vec_t* anchor_vec = reinterpret_cast<const vec_t*>(anchor + base_idx);
        const vec_t* positive_vec = reinterpret_cast<const vec_t*>(positive + base_idx);
        const vec_t* negative_vec = reinterpret_cast<const vec_t*>(negative + base_idx);

        for (int i = tid; i < vectorized_length; i += blockDim.x) {
            vec_t a_vec = __ldg(&anchor_vec[i]);
            vec_t p_vec = __ldg(&positive_vec[i]);
            vec_t n_vec = __ldg(&negative_vec[i]);
            
            double diff0 = a_vec.x - p_vec.x;
            double diff1 = a_vec.y - p_vec.y;
            local_dist_pos += diff0 * diff0 + diff1 * diff1;
            
            double diff0n = a_vec.x - n_vec.x;
            double diff1n = a_vec.y - n_vec.y;
            local_dist_neg += diff0n * diff0n + diff1n * diff1n;
        }

        int offset = vectorized_length * vec_size;
        for (int i = tid; i < remainder; i += blockDim.x) {
            int idx = base_idx + offset + i;
            double a = __ldg(&anchor[idx]);
            double p = __ldg(&positive[idx]);
            double n = __ldg(&negative[idx]);
            double diff = a - p;
            local_dist_pos += diff * diff;
            double diffn = a - n;
            local_dist_neg += diffn * diffn;
        }
    } else {
        for (int i = tid; i < feat_size; i += blockDim.x) {
            int idx = base_idx + i;
            scalar_t a = __ldg(&anchor[idx]);
            scalar_t p = __ldg(&positive[idx]);
            scalar_t n = __ldg(&negative[idx]);
            scalar_t diff = a - p;
            local_dist_pos += diff * diff;
            scalar_t diffn = a - n;
            local_dist_neg += diffn * diffn;
        }
    }

    // Warp-level reduction within each block
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        local_dist_pos += __shfl_down_sync(0xffffffff, local_dist_pos, offset);
        local_dist_neg += __shfl_down_sync(0xffffffff, local_dist_neg, offset);
    }

    __shared__ scalar_t shared_sum_pos[32];
    __shared__ scalar_t shared_sum_neg[32];

    int lane = tid % 32;
    int warp_id = tid / 32;

    if (lane == 0) {
        shared_sum_pos[warp_id] = local_dist_pos;
        shared_sum_neg[warp_id] = local_dist_neg;
    }
    __syncthreads();

    scalar_t block_sum_pos = 0;
    scalar_t block_sum_neg = 0;
    if (tid < (blockDim.x / 32)) {
        block_sum_pos = shared_sum_pos[lane];
        block_sum_neg = shared_sum_neg[lane];
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            block_sum_pos += __shfl_down_sync(0xffffffff, block_sum_pos, offset);
            block_sum_neg += __shfl_down_sync(0xffffffff, block_sum_neg, offset);
        }
        if (lane == 0) {
            scalar_t loss = sqrt(block_sum_pos) - sqrt(block_sum_neg) + margin;
            output[batch_idx] = loss < scalar_t(0) ? scalar_t(0) : loss;
        }
    }
}

// Host function to launch the kernel
torch::Tensor triplet_margin_loss_cuda(
    torch::Tensor anchor,
    torch::Tensor positive,
    torch::Tensor negative,
    float margin) {

    TORCH_CHECK(anchor.device().is_cuda(), "anchor must be a CUDA tensor");
    TORCH_CHECK(positive.device().is_cuda(), "positive must be a CUDA tensor");
    TORCH_CHECK(negative.device().is_cuda(), "negative must be a CUDA tensor");

    const int batch_size = anchor.size(0);
    const int feat_size = anchor.size(1);

    auto output = torch::zeros({batch_size}, anchor.options());
    
    // Launch one block per batch sample; use 256 threads per block
    const int threads_per_block = 256;
    const int num_blocks = batch_size;
    
    AT_DISPATCH_FLOATING_TYPES(anchor.scalar_type(), "uniform_control_flow_triplet_kernel", ([&] {
        uniform_control_flow_triplet_kernel<scalar_t><<<num_blocks, threads_per_block>>>(
            anchor.data_ptr<scalar_t>(),
            positive.data_ptr<scalar_t>(),
            negative.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            margin,
            batch_size,
            feat_size);
    }));
    
    return output.mean();
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &triplet_margin_loss_cuda, "Triplet margin loss forward (CUDA)");
}
