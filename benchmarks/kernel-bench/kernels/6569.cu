#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void optimized_max_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t outer_size,
    const int64_t dim_size,
    const int64_t inner_size
) {
    extern __shared__ __align__(sizeof(scalar_t)) unsigned char shared_mem[];
    scalar_t* shared_data = reinterpret_cast<scalar_t*>(shared_mem);
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_elements = outer_size * inner_size;
    if (idx >= total_elements) return;

    const int outer_idx = idx / inner_size;
    const int inner_idx = idx % inner_size;
    const int64_t base = outer_idx * dim_size * inner_size + inner_idx;

    scalar_t max_val = input[base];
    const int count = dim_size - 1;
    int i = 1;

    #pragma unroll
    for (; i <= count - 4; i += 4) {
        scalar_t vals[4];
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            vals[j] = input[base + (i + j) * inner_size];
        }
        
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            max_val = max(max_val, vals[j]);
        }
    }

    for (; i <= count; i++) {
        max_val = max(max_val, input[base + i * inner_size]);
    }

    shared_data[threadIdx.x] = max_val;
    __syncthreads();

    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shared_data[threadIdx.x] = max(shared_data[threadIdx.x], 
                                         shared_data[threadIdx.x + stride]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}

torch::Tensor max_reduce_cuda_forward(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();
    
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= input.size(i);
    }
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < input.dim(); i++) {
        inner_size *= input.size(i);
    }
    
    const int64_t dim_size = input.size(dim);
    
    auto output_sizes = input.sizes().vec();
    output_sizes.erase(output_sizes.begin() + dim);
    auto output = torch::empty(output_sizes, input.options());
    
    const int threads = 256;
    const int blocks = (outer_size * inner_size + threads - 1) / threads;
    const int shared_mem_size = threads * sizeof(float);
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "max_reduce_forward", ([&] {
        optimized_max_reduce_kernel<scalar_t><<<blocks, threads, shared_mem_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            dim_size,
            inner_size
        );
    }));
    
    return output;
}