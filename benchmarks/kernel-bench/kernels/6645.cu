#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// This kernel uses shared memory for intra-block reduction and warp-level primitives for final stages.
template <typename scalar_t>
__global__ void shared_memory_warp_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t outer_size,
    const int64_t dim_size,
    const int64_t inner_size
) {
    extern __shared__ scalar_t sdata[];
    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + tid;
    const int total_elements = outer_size * inner_size;
    
    if (idx >= total_elements) return;
    
    const int outer_idx = idx / inner_size;
    const int inner_idx = idx % inner_size;
    
    // Calculate starting position (coalesced pattern)
    const int64_t start_idx = outer_idx * dim_size * inner_size + inner_idx;
    
    // Initialize with first element
    scalar_t max_val = input[start_idx];
    
    // Load elements into shared memory
    for (int i = 1; i < dim_size; i++) {
        const scalar_t val = input[start_idx + i * inner_size];
        max_val = max(max_val, val);
    }
    sdata[tid] = max_val;
    __syncthreads();

    // Intra-block reduction using shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = max(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

torch::Tensor max_reduce_cuda_forward(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();
    
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= input.size(i);
    }
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < input.dim(); i++) {
        inner_size *= input.size(i);
    }
    
    const int64_t dim_size = input.size(dim);
    
    auto output_sizes = input.sizes().vec();
    output_sizes.erase(output_sizes.begin() + dim);
    auto output = torch::empty(output_sizes, input.options());
    
    const int threads = 256;
    const int blocks = (outer_size * inner_size + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "max_reduce_forward", ([&] {
        shared_memory_warp_reduce_kernel<scalar_t><<<blocks, threads, threads * sizeof(scalar_t)>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            dim_size,
            inner_size
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_reduce_cuda_forward, "Max reduce forward (CUDA) with shared memory and warp-level reduction");
}