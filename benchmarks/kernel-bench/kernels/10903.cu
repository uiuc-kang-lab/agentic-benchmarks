#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

// Device function for computing masked cumulative sum
__device__ float compute_masked_cumsum(const float* x_row, const bool* mask_row, float* output_row, int L) {
    float sum = 0;
    for (int i = 0; i < L; ++i) {
        if (mask_row[i]) {
            sum += x_row[i];
        }
        output_row[i] = sum;
    }
    return sum;
}

// CUDA kernel for masked cumulative sum
__global__ void masked_cumsum_kernel(
    const float* __restrict__ x,
    const bool* __restrict__ mask,
    float* __restrict__ output,
    int64_t N,
    int64_t L) {

    const int row = blockIdx.x;
    if (row >= N) return;

    const float* x_row = x + row * L;
    const bool* mask_row = mask + row * L;
    float* output_row = output + row * L;

    compute_masked_cumsum(x_row, mask_row, output_row, L);
}

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor masked_cumsum(
    const torch::Tensor& x,
    const torch::Tensor& mask,
    int64_t dim) {

    CHECK_INPUT(x);
    CHECK_INPUT(mask);
    TORCH_CHECK(x.sizes() == mask.sizes(), "x and mask must have the same shape");
    TORCH_CHECK(mask.scalar_type() == torch::kBool, "mask must be a boolean tensor");

    if (dim < 0) {
        dim += x.dim();
    }
    TORCH_CHECK(dim >= 0 && dim < x.dim(), "Invalid dimension");

    std::vector<int64_t> perm;
    for (int64_t i = 0; i < x.dim(); ++i) {
        if (i != dim)
            perm.push_back(i);
    }
    perm.push_back(dim);

    auto x_permuted = x.permute(perm).contiguous();
    auto mask_permuted = mask.permute(perm).contiguous();

    int64_t N = x_permuted.numel() / x_permuted.size(-1);
    int64_t L = x_permuted.size(-1);

    auto x_flat = x_permuted.view({N, L});
    auto mask_flat = mask_permuted.view({N, L});
    auto output_flat = torch::empty_like(x_flat);

    const int blocks = N;
    const int threads = BLOCK_SIZE;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "masked_cumsum_cuda", ([&] {
        masked_cumsum_kernel<<<blocks, threads>>>(
            x_flat.data_ptr<float>(),
            mask_flat.data_ptr<bool>(),
            output_flat.data_ptr<float>(),
            N,
            L
        );
    }));

    auto output_permuted = output_flat.view(x_permuted.sizes());
    std::vector<int64_t> inv_perm(perm.size());
    for (size_t i = 0; i < perm.size(); ++i) {
        inv_perm[perm[i]] = i;
    }
    auto output = output_permuted.permute(inv_perm);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &masked_cumsum, "Masked Cumulative Sum (CUDA)");
}