#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 32

// This kernel uses tiling with shared memory, ensuring that global memory accesses are coalesced
// by having threads in a warp load consecutive elements. 
__global__ void AlignedTiledMatmulKernel(const float* __restrict__ A,
                                           const float* __restrict__ B,
                                           float* __restrict__ C,
                                           int M, int K, int N) {
    __shared__ float As[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

    // Compute the row and column index for this thread
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    float sum = 0.0f;

    // Number of tiles needed to cover the K dimension
    int numTiles = (K + TILE_WIDTH - 1) / TILE_WIDTH;

    for (int t = 0; t < numTiles; t++) {
        int aCol = t * TILE_WIDTH + threadIdx.x;  // Column to load from A
        int bRow = t * TILE_WIDTH + threadIdx.y;  // Row to load from B

        // Coalesced load for A: threads in the same row load consecutive elements
        As[threadIdx.y][threadIdx.x] = (row < M && aCol < K) ? A[row * K + aCol] : 0.0f;

        // Coalesced load for B: threads in the same row (of the tile) load consecutive elements
        Bs[threadIdx.y][threadIdx.x] = (bRow < K && col < N) ? B[bRow * N + col] : 0.0f;

        __syncthreads();

        // Multiply the two tiles together
        #pragma unroll
        for (int i = 0; i < TILE_WIDTH; i++) {
            sum += As[threadIdx.y][i] * Bs[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Write the result
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

// PyTorch binding
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.is_contiguous(), "A must be contiguous");
    TORCH_CHECK(B.is_contiguous(), "B must be contiguous");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    auto C = torch::zeros({M, N}, A.options());

    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim((N + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);

    AlignedTiledMatmulKernel<<<gridDim, blockDim>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, K, N);
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Aligned tiled matrix multiplication with coalesced accesses (CUDA)");
}
