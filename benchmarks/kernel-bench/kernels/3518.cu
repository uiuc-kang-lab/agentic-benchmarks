#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Device helper: define an inline exponential function for float and double.
template <typename scalar_t>
__device__ inline scalar_t my_exp(scalar_t x);

template <>
__device__ inline float my_exp<float>(float x) {
    return expf(x);
}

template <>
__device__ inline double my_exp<double>(double x) {
    return exp(x);
}

// Combined CUDA kernel that applies the SELU activation to each element.
// It uses loop unrolling for efficiency and balanced workload distribution.
template <typename scalar_t>
__global__ void selu_kernel_combined_optimized(const scalar_t* __restrict__ input,
                                               scalar_t* __restrict__ output,
                                               size_t numel) {
    size_t total_threads = blockDim.x * gridDim.x;
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t work_per_thread = (numel + total_threads - 1) / total_threads;
    size_t start = tid * work_per_thread;
    size_t end = start + work_per_thread;
    if (end > numel) end = numel;

    // Unroll the loop to process multiple elements per thread
    #pragma unroll 4
    for (size_t i = start; i < end; i++) {
        scalar_t x = __ldg(&input[i]);
        scalar_t result = (x > static_cast<scalar_t>(0))
                              ? x
                              : static_cast<scalar_t>(1.67326324235437728481) *
                                    (my_exp(x) - static_cast<scalar_t>(1));
        output[i] = static_cast<scalar_t>(1.05070098735548049342) * result;
    }
}

// Host function that launches the combined optimized CUDA SELU kernel.
torch::Tensor selu_forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    auto output = torch::empty_like(input);
    const size_t numel = input.numel();
    const int threads = 1024;
    const int blocks = (numel + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "selu_forward_combined_optimized_cuda", ([&] {
        const scalar_t *input_ptr = input.data_ptr<scalar_t>();
        scalar_t *output_ptr = output.data_ptr<scalar_t>();
        selu_kernel_combined_optimized<scalar_t><<<blocks, threads>>>(input_ptr, output_ptr, numel);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &selu_forward, "SELU Activation Forward with Combined Optimization (CUDA)");
}