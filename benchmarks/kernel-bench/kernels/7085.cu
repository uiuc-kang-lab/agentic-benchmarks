#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <c10/cuda/CUDAStream.h>

template <typename scalar_t>
__global__ void min_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int outer,
    const int r,
    const int inner) {
    extern __shared__ char shared_mem[];
    scalar_t* shared_data = reinterpret_cast<scalar_t*>(shared_mem);
    
    const int outer_idx = blockIdx.y * blockDim.y + threadIdx.y;
    const int inner_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int block_size = blockDim.x * blockDim.y;
    
    if (outer_idx >= outer || inner_idx >= inner) return;
    
    // Starting index for reduction in the r dimension
    const int base = outer_idx * (r * inner) + inner_idx;
    scalar_t min_val = input[base];
    
    // First pass: each thread finds its local minimum
    #pragma unroll
    for (int j = 1; j < r; j++) {
        const int index = base + j * inner;
        const scalar_t curr = input[index];
        min_val = curr < min_val ? curr : min_val;
    }
    
    // Store in shared memory
    shared_data[tid] = min_val;
    __syncthreads();
    
    // Block-level reduction in shared memory
    for (int s = block_size/2; s > 0; s >>= 1) {
        if (tid < s) {
            scalar_t other = shared_data[tid + s];
            scalar_t mine = shared_data[tid];
            shared_data[tid] = other < mine ? other : mine;
        }
        __syncthreads();
    }
    
    // Write result
    if (tid == 0) {
        output[outer_idx * inner + inner_idx] = shared_data[0];
    }
}

torch::Tensor forward(torch::Tensor input, int64_t dim) {
    TORCH_CHECK(input.is_cuda(), "input must be a CUDA tensor");
    if (!input.is_contiguous()) {
        input = input.contiguous();
    }
    
    int ndim = input.dim();
    TORCH_CHECK(dim >= 0 && dim < ndim, "dim out of range");
    
    int outer = 1;
    for (int i = 0; i < dim; i++) {
        outer *= input.size(i);
    }
    int r = input.size(dim);
    int inner = 1;
    for (int i = dim + 1; i < ndim; i++) {
        inner *= input.size(i);
    }
    
    std::vector<int64_t> output_shape;
    for (int i = 0; i < ndim; i++) {
        if (i != dim) {
            output_shape.push_back(input.size(i));
        }
    }
    
    auto output = torch::empty(output_shape, input.options());
    
    // Use 2D grid and block configuration
    dim3 threads(16, 16);
    dim3 blocks(
        (inner + threads.x - 1) / threads.x,
        (outer + threads.y - 1) / threads.y
    );
    
    AT_DISPATCH_ALL_TYPES(input.scalar_type(), "min_reduce_cuda", ([&] {
        min_reduce_kernel<scalar_t><<<blocks, threads, 0, 
            c10::cuda::getCurrentCUDAStream().stream()>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer,
            r,
            inner);
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Min reduction over a specified dimension (CUDA)");
}