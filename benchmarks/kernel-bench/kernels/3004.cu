#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel with atomic operations for race condition handling
__global__ void tanh_atomic_kernel(const float* __restrict__ input,
                                    float* __restrict__ output,
                                    const int size) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Shared memory for reducing global memory atomic operations
    __shared__ float shared_output[512];

    for (int i = idx; i < size; i += stride) {
        float val = tanhf(input[i]);
        atomicAdd(&shared_output[threadIdx.x], val);
    }
    __syncthreads();

    // Atomic add to global memory from shared memory
    if (threadIdx.x == 0) {
        for (int i = 0; i < blockDim.x; ++i) {
            atomicAdd(&output[blockIdx.x], shared_output[i]);
        }
    }
}

// Forward function exposed to Python
torch::Tensor forward(torch::Tensor input) {
    auto output = torch::zeros_like(input);
    const int numel = input.numel();
    const int threads = 512;
    const int blocks = (numel + threads - 1) / threads;

    tanh_atomic_kernel<<<blocks, threads>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        numel);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Tanh with atomic operations (CUDA)");
}