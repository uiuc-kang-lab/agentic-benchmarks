#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void hybrid_avg_pool1d_kernel(
    const float *input,
    float *output,
    int kernel_size,
    int stride,
    int padding,
    int input_length,
    int output_length,
    int batch_size,
    int in_channels) {

    extern __shared__ float shared_input[];
    
    int o_start = blockIdx.x * blockDim.x * 2 + threadIdx.x * 2;
    int channel = blockIdx.y;
    int batch = blockIdx.z;

    if (channel >= in_channels || batch >= batch_size) return;

    int input_offset = batch * in_channels * input_length + channel * input_length;
    int output_offset = batch * in_channels * output_length + channel * output_length;

    for (int i = 0; i < 2; i++) {
        int o = o_start + i;
        if (o >= output_length) break;

        float sum = 0.0f;
        
        for (int k = 0; k < kernel_size; ++k) {
            int pos_padded = o * stride + k;
            int pos_input = pos_padded - padding;
            
            if (pos_input >= 0 && pos_input < input_length) {
                shared_input[threadIdx.x * kernel_size + k] = __ldg(&input[input_offset + pos_input]);
            } else {
                shared_input[threadIdx.x * kernel_size + k] = 0.0f;
            }
        }

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < kernel_size; ++k) {
            sum += shared_input[threadIdx.x * kernel_size + k];
        }

        output[output_offset + o] = sum / kernel_size;
    }
}

torch::Tensor hybrid_avg_pool1d_forward(
    const torch::Tensor &x,
    int kernel_size,
    int stride,
    int padding) {
    
    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(kernel_size > 0 && stride > 0 && padding >= 0, "Invalid kernel parameters");

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int input_length = x.size(2);
    int output_length = (input_length + 2 * padding - kernel_size) / stride + 1;

    auto output = torch::empty({batch_size, in_channels, output_length}, x.options());

    dim3 threads(128);
    dim3 grid(
        (output_length + threads.x * 2 - 1) / (threads.x * 2),
        in_channels,
        batch_size
    );

    size_t shared_memory_size = threads.x * kernel_size * sizeof(float);

    hybrid_avg_pool1d_kernel<<<grid, threads, shared_memory_size>>>(
        x.data_ptr<float>(),
        output.data_ptr<float>(),
        kernel_size,
        stride,
        padding,
        input_length,
        output_length,
        batch_size,
        in_channels
    );

    return output;
}