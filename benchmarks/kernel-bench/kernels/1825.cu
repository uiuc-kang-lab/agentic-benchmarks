#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 128
#define NUM_STREAMS 4

__global__ void triangular_mm_kernel_stride(const float* __restrict__ A,
                                             const float* __restrict__ B,
                                             float* __restrict__ C,
                                             int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loops to handle large workloads
    for (int r = row; r < N; r += blockDim.y * gridDim.y) {
        for (int c = col; c < N; c += blockDim.x * gridDim.x) {
            if (r < N && c < N) {
                float sum = 0.f;
                for (int k = c; k <= r; ++k) {
                    sum += __ldg(&A[r * N + k]) * __ldg(&B[k * N + c]);
                }
                C[r * N + c] = (r >= c) ? sum : 0.f;
            }
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threads - 1) / threads, (N + threads - 1) / threads);

    // Launch the CUDA kernel with stride loops
    triangular_mm_kernel_stride<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}