#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32
#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void matmul_kernel_2d(const float* __restrict__ A,
                                const float* __restrict__ B,
                                float* __restrict__ C,
                                const int M, const int N, const int K) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    // Block row and column
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    
    // Thread row and column within tile
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    // Row and column indices for C
    const int row = by * TILE_SIZE + ty;
    const int col = bx * TILE_SIZE + tx;

    // Initialize accumulator
    float sum = 0.0f;

    // Loop over tiles
    for (int tile = 0; tile < (K + TILE_SIZE - 1) / TILE_SIZE; ++tile) {
        // Load tiles into shared memory
        if (row < M && tile * TILE_SIZE + tx < K) {
            As[ty][tx] = A[row * K + tile * TILE_SIZE + tx];
        } else {
            As[ty][tx] = 0.0f;
        }

        if (tile * TILE_SIZE + ty < K && col < N) {
            Bs[ty][tx] = B[(tile * TILE_SIZE + ty) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }

        __syncthreads();

        // Compute partial dot product for this tile
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();
    }

    // Write result
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

void matrix_multiply_cuda(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);

    const int M = A.size(0);
    const int K = A.size(1);
    const int N = B.size(1);

    const float* d_A = A.data_ptr<float>();
    const float* d_B = B.data_ptr<float>();
    float* d_C = C.data_ptr<float>();

    // Configure kernel launch parameters
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks(
        (N + TILE_SIZE - 1) / TILE_SIZE,
        (M + TILE_SIZE - 1) / TILE_SIZE
    );

    // Launch kernel with computed dimensions
    matmul_kernel_2d<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    const int M = A.size(0);
    const int N = B.size(1);

    auto options = torch::TensorOptions()
        .dtype(A.dtype())
        .device(A.device())
        .requires_grad(false);
    
    torch::Tensor C = torch::empty({M, N}, options);
    matrix_multiply_cuda(A, B, C);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized tiled matrix multiplication (CUDA)");
}