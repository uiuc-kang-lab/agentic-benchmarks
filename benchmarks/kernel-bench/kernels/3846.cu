#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_softplus(scalar_t x) {
    if (x > 20.0) {
        return x;
    } else if (x < -20.0) {
        return exp(x);
    } else {
        const scalar_t exp_x = exp(x);
        return log1p(exp_x);
    }
}

template <typename scalar_t>
__global__ void softplus_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < size) {
        // Use __ldg for read-only access with texture cache optimization
        const scalar_t x = __ldg(&input[idx]);
        output[idx] = compute_softplus(x);
    }
}

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    // Use 256 threads (multiple of 32) for better memory coalescing
    const int threads = 256;
    const int blocks = (size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "softplus_forward_cuda", ([&] {
        softplus_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}