#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N) {
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            float sum = 0.f;
            // Process elements in chunks of warp size
            const unsigned int FULL_MASK = 0xffffffff;
            const int WARP_SIZE = 32;
            const int warp_id = threadIdx.x / WARP_SIZE;
            const int lane_id = threadIdx.x % WARP_SIZE;
            
            for (int k = col + lane_id; k <= row; k += WARP_SIZE) {
                if (k <= row) {
                    sum += A[row * N + k] * B[k * N + col];
                }
            }
            
            // Warp-level reduction using shuffle
            #pragma unroll
            for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
                sum += __shfl_down_sync(FULL_MASK, sum, offset);
            }
            
            // Only the first thread in the warp writes the result
            if (lane_id == 0) {
                C[row * N + col] = sum;
            }
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Adjust block dimensions to align with warp size
    const int WARP_SIZE = 32;
    dim3 threadsPerBlock(WARP_SIZE, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}