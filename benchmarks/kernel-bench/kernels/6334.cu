#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel using warp-level primitives with loop unrolling for reduction across the reduce dimension.
// Each warp computes one output element by partitioning the reduction workload among its threads.

template <typename scalar_t>
__global__ void unroll_warp_reduce_sum_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t reduce_size,
    int64_t inner_size,
    int64_t total_outputs) {

    // Calculate global warp id and lane id
    const int warpSize = 32;
    int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = global_thread_id / warpSize;
    int lane = global_thread_id % warpSize;

    // Total warps available
    int total_warps = (gridDim.x * blockDim.x) / warpSize;

    // Each warp processes one output element in a grid-stride loop over warps
    for (int out_idx = warp_id; out_idx < total_outputs; out_idx += total_warps) {
        // Map the output index to the corresponding outer and inner indices
        int outer_idx = out_idx / inner_size;
        int inner_idx = out_idx % inner_size;

        // Compute the base address for the reduction
        int64_t base = outer_idx * reduce_size * inner_size + inner_idx;
        scalar_t sum_val = 0;

        // Each thread in the warp accumulates a partial sum over the reduction dimension, striding by warpSize
        #pragma unroll
        for (int i = lane; i < reduce_size; i += warpSize) {
            sum_val += input[base + i * inner_size];
        }

        // Perform warp-level reduction using shuffle down
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            sum_val += __shfl_down_sync(0xFFFFFFFF, sum_val, offset);
        }

        // Lane 0 writes the final result for this output element
        if (lane == 0) {
            output[out_idx] = sum_val;
        }
    }
}

// CUDA wrapper function
torch::Tensor sum_reduce_cuda(torch::Tensor input, int64_t dim) {
    // Adjust for negative dimensions
    if (dim < 0) dim += input.dim();

    auto sizes = input.sizes().vec();
    int64_t reduce_size = sizes[dim];

    // Compute outer and inner dimensions
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }
    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }

    // Output tensor: replacing reduction dimension with 1
    sizes[dim] = 1;
    auto output = torch::empty(sizes, input.options());

    // Total number of output elements is outer_size x inner_size
    int64_t total_outputs = outer_size * inner_size;

    // Configure kernel launch parameters using warp-level reduction
    // Each output element is computed by one warp (32 threads)
    const int warpSize = 32;
    int required_warps = total_outputs;      // one warp per output element
    int total_threads = required_warps * warpSize;
    int threads = 256;                       // Choose block size as a multiple of 32 (e.g., 256 threads per block)
    int blocks = (total_threads + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sum_reduce_cuda", ([&] {
        unroll_warp_reduce_sum_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            reduce_size,
            inner_size,
            total_outputs
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &sum_reduce_cuda, "Sum reduction forward (CUDA)");
}
