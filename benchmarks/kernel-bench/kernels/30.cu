#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAException.h>

#define TILE_SIZE 32
#define BM 4
#define BN 8
#define KT 32

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CHECK_FLOAT(x) TORCH_CHECK(x.scalar_type() == torch::kFloat32, #x " must be a float32 tensor")

__global__ void optimized_matmul_kernel(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;

    if (row >= N || col >= N) return;

    float C_value = 0.0f;

    for (int m = 0; m < (N + TILE_SIZE - 1) / TILE_SIZE; ++m) {
        if (m * TILE_SIZE + tx < N) {
            As[ty][tx] = A[row * N + m * TILE_SIZE + tx];
        } else {
            As[ty][tx] = 0.0f;
        }

        if (m * TILE_SIZE + ty < N) {
            Bs[ty][tx] = B[(m * TILE_SIZE + ty) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }

        __syncthreads();

        float partial = As[ty][tx] * Bs[ty][tx];

        // Warp-level reduction
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            partial += __shfl_down_sync(0xFFFFFFFF, partial, offset);
        }
        
        if(tx % warpSize == 0) {
            C_value += partial;
        }

        __syncthreads();
    }

    if(tx % warpSize == 0) {
        C[row * N + col] = C_value;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_FLOAT(A);
    CHECK_FLOAT(B);

    TORCH_CHECK(A.dim() == 2 && A.size(0) == A.size(1), "A must be a square matrix");
    TORCH_CHECK(B.dim() == 2 && B.size(0) == B.size(1), "B must be a square matrix");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be of the same size");

    int64_t N = A.size(0);
    auto C = torch::zeros({N, N}, A.options());

    const float* A_data = A.data_ptr<float>();
    const float* B_data = B.data_ptr<float>();
    float* C_data = C.data_ptr<float>();

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    optimized_matmul_kernel<<<blocksPerGrid, threadsPerBlock>>>(A_data, B_data, C_data, N);

    C10_CUDA_CHECK(hipGetLastError());

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized matrix multiplication kernel (CUDA)");
}