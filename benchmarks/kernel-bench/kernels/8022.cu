#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK((x).is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK((x).is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define MAX_WEIGHT_SIZE 4096

// Use constant memory for weights as a cache to speed-up accesses
__constant__ float const_weight[MAX_WEIGHT_SIZE];

// Modular device functions for index calculations
__device__ inline int get_input_index(int b, int c, int i, int in_channels, int input_width) {
    return b * in_channels * input_width + c * input_width + i;
}

__device__ inline int get_output_index(int b, int o, int j, int out_channels, int output_width) {
    return b * out_channels * output_width + o * output_width + j;
}

__global__ void optimized_transposed_conv1d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int input_width,
    int output_width,
    int kernel_size,
    int stride,
    int padding,
    int groups) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch_size * out_channels * output_width;
    if (index >= total) return;

    int j = index % output_width;
    int o = (index / output_width) % out_channels;
    int b = index / (output_width * out_channels);

    int group_in_channels = in_channels / groups;
    int group_size_out = out_channels / groups;
    int g = o / group_size_out;
    int c_start = g * group_in_channels;

    float sum = 0.0f;
    for (int k = 0; k < kernel_size; ++k) {
        int i = j + padding - k;
        if (i % stride != 0) continue;
        i /= stride;
        if (i < 0 || i >= input_width) continue;

        for (int ic = 0; ic < group_in_channels; ++ic) {
            int input_idx = get_input_index(b, c_start + ic, i, in_channels, input_width);
            int weight_idx = ((ic * group_size_out + (o - g * group_size_out)) * kernel_size + k);
            sum += input[input_idx] * const_weight[weight_idx];
        }
    }

    if (bias != nullptr) {
        sum += bias[o];
    }

    int out_idx = get_output_index(b, o, j, out_channels, output_width);
    output[out_idx] = sum;
}

// Host wrapper function
torch::Tensor forward(
    const torch::Tensor& x,
    const torch::Tensor& weight,
    const c10::optional<torch::Tensor>& bias,
    int64_t stride,
    int64_t padding,
    int64_t output_padding,
    int64_t groups) {

    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int input_width = x.size(2);
    int kernel_size = weight.size(2);
    int group_size_out = weight.size(1);
    int out_channels = group_size_out * groups;
    int output_width = (input_width - 1) * stride - 2 * padding + kernel_size + output_padding;

    auto output = torch::zeros({batch_size, out_channels, output_width}, x.options());

    int num_weight_elems = weight.numel();
    TORCH_CHECK(num_weight_elems <= MAX_WEIGHT_SIZE, "Weight size exceeds constant memory limit");
    hipMemcpyToSymbol(HIP_SYMBOL(const_weight), weight.data_ptr<float>(), num_weight_elems * sizeof(float), 0, hipMemcpyDeviceToDevice);

    int total_threads = batch_size * out_channels * output_width;
    int threads = 256;
    int blocks = (total_threads + threads - 1) / threads;

    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        bias_ptr = bias.value().data_ptr<float>();
    }

    optimized_transposed_conv1d_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        x.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        out_channels,
        input_width,
        output_width,
        kernel_size,
        stride,
        padding,
        groups
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Transposed 1D convolution forward (CUDA) using constant memory and efficient indexing");
}
