#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

namespace py = pybind11;

// Define tile dimensions for spatial workload distribution
#define TILE_W 16
#define TILE_H 16

// Utility function to parse an int or sequence of ints from a Python object
inline std::vector<int64_t> parseIntArrayRef(const py::object &obj) {
    std::vector<int64_t> result;
    if (py::isinstance<py::int_>(obj)) {
        result.push_back(obj.cast<int64_t>());
    } else if (py::isinstance<py::sequence>(obj)) {
        for (auto item : obj.cast<py::sequence>()) {
            result.push_back(py::cast<int64_t>(item));
        }
    } else {
        throw std::runtime_error("Expected int or sequence of ints");
    }
    return result;
}

// Optimized CUDA kernel for ConvTranspose2d with even workload distribution
// We use a 3D grid: gridDim.z maps (batch, out_channel) and a 2D block for spatial tiling
__global__ void conv_transpose2d_even_workload_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,  // may be nullptr
    float* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int h_in,
    const int w_in,
    const int out_channels,
    const int h_out,
    const int w_out,
    const int kernel_size,
    const int stride_h,
    const int stride_w,
    const int padding_h,
    const int padding_w,
    const int groups,
    const int in_channels_per_group,
    const int out_channels_per_group
) {
    // gridDim.z covers combined (batch, out_channel)
    int n_c = blockIdx.z;
    int n = n_c / out_channels;
    int c = n_c % out_channels;

    // Each block covers a TILE_W x TILE_H output tile
    int out_w = blockIdx.x * TILE_W + threadIdx.x;
    int out_h = blockIdx.y * TILE_H + threadIdx.y;
    if (out_w >= w_out || out_h >= h_out)
        return;

    int g = c / out_channels_per_group;
    int c_local = c % out_channels_per_group;
    float sum = 0.0f;

    // If kernel size is 3, unroll loops for faster execution
    if (kernel_size == 3) {
        #pragma unroll
        for (int kh = 0; kh < 3; ++kh) {
            int h_in_candidate = out_h + padding_h - kh;
            if (h_in_candidate % stride_h != 0) continue;
            int h_in_idx = h_in_candidate / stride_h;
            if (h_in_idx < 0 || h_in_idx >= h_in) continue;
            // Unroll kw = 0, 1, 2
            {   // kw = 0
                int kw = 0;
                int w_in_candidate = out_w + padding_w - kw;
                if (w_in_candidate % stride_w == 0) {
                    int w_in_idx = w_in_candidate / stride_w;
                    if (w_in_idx >= 0 && w_in_idx < w_in) {
                        for (int r = 0; r < in_channels_per_group; ++r) {
                            int in_channel = g * in_channels_per_group + r;
                            int input_idx = ((n * in_channels + in_channel) * h_in + h_in_idx) * w_in + w_in_idx;
                            int weight_idx = (((g * in_channels_per_group + r) * out_channels_per_group + c_local) * kernel_size + kh) * kernel_size + kw;
                            sum += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
                        }
                    }
                }
            }
            {   // kw = 1
                int kw = 1;
                int w_in_candidate = out_w + padding_w - kw;
                if (w_in_candidate % stride_w == 0) {
                    int w_in_idx = w_in_candidate / stride_w;
                    if (w_in_idx >= 0 && w_in_idx < w_in) {
                        for (int r = 0; r < in_channels_per_group; ++r) {
                            int in_channel = g * in_channels_per_group + r;
                            int input_idx = ((n * in_channels + in_channel) * h_in + h_in_idx) * w_in + w_in_idx;
                            int weight_idx = (((g * in_channels_per_group + r) * out_channels_per_group + c_local) * kernel_size + kh) * kernel_size + kw;
                            sum += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
                        }
                    }
                }
            }
            {   // kw = 2
                int kw = 2;
                int w_in_candidate = out_w + padding_w - kw;
                if (w_in_candidate % stride_w == 0) {
                    int w_in_idx = w_in_candidate / stride_w;
                    if (w_in_idx >= 0 && w_in_idx < w_in) {
                        for (int r = 0; r < in_channels_per_group; ++r) {
                            int in_channel = g * in_channels_per_group + r;
                            int input_idx = ((n * in_channels + in_channel) * h_in + h_in_idx) * w_in + w_in_idx;
                            int weight_idx = (((g * in_channels_per_group + r) * out_channels_per_group + c_local) * kernel_size + kh) * kernel_size + kw;
                            sum += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
                        }
                    }
                }
            }
        }
    } else {
        // Generic loop for arbitrary kernel sizes
        for (int kh = 0; kh < kernel_size; ++kh) {
            int h_in_candidate = out_h + padding_h - kh;
            if (h_in_candidate % stride_h != 0) continue;
            int h_in_idx = h_in_candidate / stride_h;
            if (h_in_idx < 0 || h_in_idx >= h_in) continue;
            for (int kw = 0; kw < kernel_size; ++kw) {
                int w_in_candidate = out_w + padding_w - kw;
                if (w_in_candidate % stride_w != 0) continue;
                int w_in_idx = w_in_candidate / stride_w;
                if (w_in_idx < 0 || w_in_idx >= w_in) continue;
                for (int r = 0; r < in_channels_per_group; ++r) {
                    int in_channel = g * in_channels_per_group + r;
                    int input_idx = ((n * in_channels + in_channel) * h_in + h_in_idx) * w_in + w_in_idx;
                    int weight_idx = (((g * in_channels_per_group + r) * out_channels_per_group + c_local) * kernel_size + kh) * kernel_size + kw;
                    sum += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
                }
            }
        }
    }

    if (bias != nullptr) {
        sum += __ldg(&bias[c]);
    }

    int output_idx = ((n * out_channels + c) * h_out + out_h) * w_out + out_w;
    output[output_idx] = sum;
}

// Forward function to set up dimensions and launch the kernel
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    py::object stride = py::int_(1),
    py::object padding = py::int_(0),
    py::object output_padding = py::int_(0),
    int64_t groups = 1
) {
    auto stride_vec = parseIntArrayRef(stride);
    auto padding_vec = parseIntArrayRef(padding);
    auto output_padding_vec = parseIntArrayRef(output_padding);

    int stride_h = stride_vec[0];
    int stride_w = (stride_vec.size() > 1) ? stride_vec[1] : stride_h;
    int padding_h = padding_vec[0];
    int padding_w = (padding_vec.size() > 1) ? padding_vec[1] : padding_h;
    int output_padding_h = output_padding_vec[0];
    int output_padding_w = (output_padding_vec.size() > 1) ? output_padding_vec[1] : output_padding_h;

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int h_in = x.size(2);
    const int w_in = x.size(3);
    const int kernel_size = weight.size(2);  // assuming square kernel
    int out_channels = weight.size(1) * groups;

    // Calculate output dimensions
    int h_out = (h_in - 1) * stride_h - 2 * padding_h + kernel_size + output_padding_h;
    int w_out = (w_in - 1) * stride_w - 2 * padding_w + kernel_size + output_padding_w;

    auto output_tensor = torch::zeros({batch_size, out_channels, h_out, w_out}, x.options());

    int in_channels_per_group = in_channels / groups;
    int out_channels_per_group = out_channels / groups;

    // Set up 3D grid: x and y for spatial dimensions, z for combined (batch, channel)
    dim3 blockDim(TILE_W, TILE_H, 1);
    dim3 gridDim((w_out + TILE_W - 1) / TILE_W,
                 (h_out + TILE_H - 1) / TILE_H,
                 batch_size * out_channels);

    const float* input_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float* output_ptr = output_tensor.data_ptr<float>();

    conv_transpose2d_even_workload_kernel<<<gridDim, blockDim>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch_size,
        in_channels,
        h_in,
        w_in,
        out_channels,
        h_out,
        w_out,
        kernel_size,
        stride_h,
        stride_w,
        padding_h,
        padding_w,
        groups,
        in_channels_per_group,
        out_channels_per_group
    );

    hipDeviceSynchronize();
    return output_tensor;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Even workload distribution for ConvTranspose2d forward kernel",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride") = 1,
          py::arg("padding") = 0,
          py::arg("output_padding") = 0,
          py::arg("groups") = 1);
}
