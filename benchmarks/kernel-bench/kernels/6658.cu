#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void unrolled_coalesced_max_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t outer_size,
    const int64_t dim_size,
    const int64_t inner_size
) {
    int outer_idx = blockIdx.x;
    int inner_idx = blockIdx.y * blockDim.x + threadIdx.x;
    if (inner_idx >= inner_size) return;

    const int64_t base_offset = outer_idx * dim_size * inner_size + inner_idx;
    scalar_t max_val = __ldg(input + base_offset);

    int i = 1;
    #pragma unroll 8
    for (; i <= dim_size - 8; i += 8) {
        scalar_t v1 = __ldg(input + base_offset + i*inner_size);
        scalar_t v2 = __ldg(input + base_offset + (i+1)*inner_size);
        scalar_t v3 = __ldg(input + base_offset + (i+2)*inner_size);
        scalar_t v4 = __ldg(input + base_offset + (i+3)*inner_size);
        scalar_t v5 = __ldg(input + base_offset + (i+4)*inner_size);
        scalar_t v6 = __ldg(input + base_offset + (i+5)*inner_size);
        scalar_t v7 = __ldg(input + base_offset + (i+6)*inner_size);
        scalar_t v8 = __ldg(input + base_offset + (i+7)*inner_size);
        scalar_t max1 = max(max(max(v1, v2), max(v3, v4)), max(max(v5, v6), max(v7, v8)));
        max_val = max(max_val, max1);
    }
    
    for (; i < dim_size; i++) {
        max_val = max(max_val, __ldg(input + base_offset + i*inner_size));
    }

    output[outer_idx * inner_size + inner_idx] = max_val;
}

torch::Tensor max_reduce_cuda_forward(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();

    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) outer_size *= input.size(i);

    int64_t inner_size = 1;
    for (int i = dim + 1; i < input.dim(); i++) inner_size *= input.size(i);

    auto output_sizes = input.sizes().vec();
    output_sizes.erase(output_sizes.begin() + dim);
    auto output = torch::empty(output_sizes, input.options());

    const int threads = 256;
    const int blocks_y = (inner_size + threads - 1) / threads;
    dim3 grid(outer_size, blocks_y);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "max_reduce_forward", ([&] {
        unrolled_coalesced_max_reduce_kernel<scalar_t><<<grid, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            input.size(dim),
            inner_size
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_reduce_cuda_forward, "Unrolled coalesced max reduction (CUDA)");
}