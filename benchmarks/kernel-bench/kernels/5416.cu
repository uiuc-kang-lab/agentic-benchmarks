#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* input,
    scalar_t* output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    // 2D thread indexing
    const int ox = blockIdx.x * blockDim.x + threadIdx.x;
    const int oy = blockIdx.y * blockDim.y + threadIdx.y;
    const int c = blockIdx.z % channels;
    const int b = blockIdx.z / channels;

    if (ox >= output_width || oy >= output_height) return;

    // Calculate output index
    const int output_idx = ((b * channels + c) * output_height + oy) * output_width + ox;
    
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    #pragma unroll
    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = oy * stride - padding + kh * dilation;
        if (ih >= 0 && ih < input_height) {
            #pragma unroll
            for (int kw = 0; kw < kernel_size; kw++) {
                const int iw = ox * stride - padding + kw * dilation;
                if (iw >= 0 && iw < input_width) {
                    const int input_idx = ((b * channels + c) * input_height + ih) * input_width + iw;
                    max_val = fmaxf(max_val, input[input_idx]);
                }
            }
        }
    }

    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    // Optimized block configuration
    dim3 threads(32, 32); // Adjusted block size for better performance
    dim3 blocks(
        (output_width + threads.x - 1) / threads.x,
        (output_height + threads.y - 1) / threads.y,
        batch_size * channels
    );

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}
