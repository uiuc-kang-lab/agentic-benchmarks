#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Device function for GELU activation computation
__device__ float compute_gelu(float x) {
    const float sqrt_2_over_pi = 0.7978845608f;
    const float coeff = 0.044715f;
    float x_cubed = x * x * x;
    float inner = x + coeff * x_cubed;
    inner *= sqrt_2_over_pi;
    float tanh_val = tanhf(inner);
    return 0.5f * x * (1.0f + tanh_val);
}

// Kernel that applies the GELU activation
__global__ void gelu_kernel(const float* __restrict__ x, float* __restrict__ y, int n) {
    int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        int element_index = idx + i * blockDim.x;
        if (element_index < n) {
            y[element_index] = compute_gelu(x[element_index]);
        }
    }
}

// Torch binding to launch GELU kernel
torch::Tensor gelu_forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(x.is_contiguous(), "Input tensor must be contiguous");
    
    auto y = torch::empty_like(x);
    int n = x.numel();
    
    const int threads = 256;
    int blocks = (n + threads * 4 - 1) / (threads * 4);
    
    gelu_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        n
    );
    
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &gelu_forward, "Modularized GELU forward CUDA implementation");
}