#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Templated kernel that performs hinge loss computation with reduction
// using a grid-stride loop and shared memory reduction.

template <int BLOCK_SIZE>
__global__ void hinge_loss_tuned_kernel(const float* __restrict__ predictions,
                                         const float* __restrict__ targets,
                                         float* __restrict__ partialSums,
                                         int n) {
    __shared__ float shared_data[BLOCK_SIZE];
    int tid = threadIdx.x;
    int idx = blockIdx.x * BLOCK_SIZE + tid;
    float localSum = 0.0f;
    
    // Grid-stride loop: allow each thread to process multiple elements
    for (int i = idx; i < n; i += gridDim.x * BLOCK_SIZE) {
        float pred = __ldg(&predictions[i]);
        float targ = __ldg(&targets[i]);
        localSum += fmaxf(0.0f, 1.0f - pred * targ);
    }
    
    shared_data[tid] = localSum;
    __syncthreads();
    
    // In-block reduction in shared memory
    for (int stride = BLOCK_SIZE / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }
    
    // Write the result of this block to global memory
    if (tid == 0) {
        partialSums[blockIdx.x] = shared_data[0];
    }
}

// The forward function selects the block size based on the problem size
// and launches the appropriate templated kernel specialization.

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    CHECK_INPUT(predictions);
    CHECK_INPUT(targets);
    
    int n = predictions.numel();

    // Choose block size based on problem size to experiment with different configurations
    int block_size;
    if (n < 512) {
        block_size = 32;
    } else if (n < 4096) {
        block_size = 64;
    } else if (n < 100000) {
        block_size = 128;
    } else if (n < 10000000) {
        block_size = 256;
    } else {
        block_size = 512;
    }
    
    int blocks = (n + block_size - 1) / block_size;
    auto partialSums = torch::empty({blocks}, predictions.options());
    
    // Launch the kernel with the chosen block size
    switch(block_size) {
        case 32:
            hinge_loss_tuned_kernel<32><<<blocks, 32>>>(predictions.data_ptr<float>(),
                                                          targets.data_ptr<float>(),
                                                          partialSums.data_ptr<float>(),
                                                          n);
            break;
        case 64:
            hinge_loss_tuned_kernel<64><<<blocks, 64>>>(predictions.data_ptr<float>(),
                                                          targets.data_ptr<float>(),
                                                          partialSums.data_ptr<float>(),
                                                          n);
            break;
        case 128:
            hinge_loss_tuned_kernel<128><<<blocks, 128>>>(predictions.data_ptr<float>(),
                                                          targets.data_ptr<float>(),
                                                          partialSums.data_ptr<float>(),
                                                          n);
            break;
        case 256:
            hinge_loss_tuned_kernel<256><<<blocks, 256>>>(predictions.data_ptr<float>(),
                                                          targets.data_ptr<float>(),
                                                          partialSums.data_ptr<float>(),
                                                          n);
            break;
        case 512:
            hinge_loss_tuned_kernel<512><<<blocks, 512>>>(predictions.data_ptr<float>(),
                                                          targets.data_ptr<float>(),
                                                          partialSums.data_ptr<float>(),
                                                          n);
            break;
        default:
            hinge_loss_tuned_kernel<256><<<blocks, 256>>>(predictions.data_ptr<float>(),
                                                          targets.data_ptr<float>(),
                                                          partialSums.data_ptr<float>(),
                                                          n);
            break;
    }
    
    // Final reduction: compute the mean hinge loss on the GPU
    return torch::sum(partialSums) / n;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Tuned Hinge Loss Forward");
}
