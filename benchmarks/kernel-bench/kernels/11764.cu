#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

constexpr int WARP_SIZE = 32;

__global__ void tuned_block_kl_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets,
    float* __restrict__ output,
    const int n) {
    
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int vec_size = 4;
    const int vec_count = n / vec_size;

    float sum = 0.0f;

    // Vectorized processing with 128-bit loads
    for (int vec_idx = tid; vec_idx < vec_count; vec_idx += stride) {
        const float4 log_vec = *reinterpret_cast<const float4*>(log_predictions + vec_idx * vec_size);
        const float4 tgt_vec = *reinterpret_cast<const float4*>(targets + vec_idx * vec_size);
        
        sum += expf(log_vec.x) - tgt_vec.x * log_vec.x;
        sum += expf(log_vec.y) - tgt_vec.y * log_vec.y;
        sum += expf(log_vec.z) - tgt_vec.z * log_vec.z;
        sum += expf(log_vec.w) - tgt_vec.w * log_vec.w;
    }

    // Process remaining elements
    const int scalar_base = vec_count * vec_size;
    for (int i = scalar_base + tid; i < n; i += stride) {
        sum += expf(log_predictions[i]) - targets[i] * log_predictions[i];
    }

    // Warp reduction with shuffle instructions
    for (int offset = WARP_SIZE/2; offset > 0; offset >>= 1)
        sum += __shfl_down_sync(0xffffffff, sum, offset);

    // Block-level reduction
    if (threadIdx.x % WARP_SIZE == 0) {
        atomicAdd(output, sum);
    }
}

torch::Tensor tuned_block_kl_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());

    // Optimized block configuration for H100
    const int block_size = 512;  // 16 warps/block
    const int grid_size = std::min((n + block_size - 1) / block_size, 512);

    tuned_block_kl_kernel<<<grid_size, block_size>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &tuned_block_kl_forward, "KL divergence with tuned block size (CUDA)");
}