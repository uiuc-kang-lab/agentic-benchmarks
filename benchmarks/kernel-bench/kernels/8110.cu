#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Helper function for warp-level reduction
__inline__ __device__ float warpReduceSum(float val) {
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

// This kernel computes one output element per block with unrolled loops
__global__ void conv_transposed_1d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_length,
    int out_length,
    int kernel_size,
    int stride,
    int padding,
    int output_padding,
    int groups
) {
    int index = blockIdx.x;
    int total_output = batch_size * out_channels * out_length / blockDim.x;
    if (index >= total_output) return;

    int out_x = index % out_length;
    int c_out = (index / out_length) % out_channels;
    int n = index / (out_length * out_channels);

    int out_channels_per_group = out_channels / groups;
    int in_channels_per_group = in_channels / groups;
    int group = c_out / out_channels_per_group;
    int c_out_local = c_out % out_channels_per_group;

    float partial = 0.0f;
    
    // Compute the number of full unrolls (4 iterations per unroll)
    const int UNROLL_FACTOR = 4;
    int total_iters = in_channels_per_group * kernel_size;
    int unrolled_iters = (total_iters / UNROLL_FACTOR) * UNROLL_FACTOR;
    
    // Handle full unrolls
    #pragma unroll
    for (int idx = threadIdx.x; idx < unrolled_iters; idx += blockDim.x * UNROLL_FACTOR) {
        #pragma unroll
        for (int u = 0; u < UNROLL_FACTOR; u++) {
            int current_idx = idx + u * blockDim.x;
            int channel_local = current_idx / kernel_size;
            int k = current_idx % kernel_size;
            int in_channel = group * in_channels_per_group + channel_local;

            int shifted = out_x + padding - k;
            if (shifted % stride == 0) {
                int in_x = shifted / stride;
                if (in_x >= 0 && in_x < in_length) {
                    int input_idx = n * (in_channels * in_length) + in_channel * in_length + in_x;
                    int weight_idx = in_channel * (out_channels_per_group * kernel_size) + 
                                   c_out_local * kernel_size + k;
                    partial += input[input_idx] * weight[weight_idx];
                }
            }
        }
    }

    // Handle remaining iterations
    #pragma unroll
    for (int idx = threadIdx.x + unrolled_iters; idx < total_iters; idx += blockDim.x) {
        int channel_local = idx / kernel_size;
        int k = idx % kernel_size;
        int in_channel = group * in_channels_per_group + channel_local;

        int shifted = out_x + padding - k;
        if (shifted % stride == 0) {
            int in_x = shifted / stride;
            if (in_x >= 0 && in_x < in_length) {
                int input_idx = n * (in_channels * in_length) + in_channel * in_length + in_x;
                int weight_idx = in_channel * (out_channels_per_group * kernel_size) + 
                               c_out_local * kernel_size + k;
                partial += input[input_idx] * weight[weight_idx];
            }
        }
    }

    partial = warpReduceSum(partial);

    if (threadIdx.x == 0) {
        float bias_val = 0.0f;
        if (bias != nullptr) {
            bias_val = bias[c_out];
        }
        output[index] = partial + bias_val;
    }
}

torch::Tensor forward(
    const torch::Tensor& input,
    const torch::Tensor& weight,
    const c10::optional<torch::Tensor>& bias,
    int64_t stride,
    int64_t padding,
    int64_t output_padding,
    int64_t groups
) {
    CHECK_INPUT(input);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    int batch_size = input.size(0);
    int in_channels = input.size(1);
    int in_length = input.size(2);
    int kernel_size = weight.size(2);
    int out_channels_per_group = weight.size(1);
    int out_channels = out_channels_per_group * groups;
    int out_length = (in_length - 1) * stride - 2 * padding + kernel_size + output_padding;

    auto output_tensor = torch::zeros({batch_size, out_channels, out_length}, input.options());

    const float* input_ptr = input.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = (bias.has_value() ? bias.value().data_ptr<float>() : nullptr);
    float* output_ptr = output_tensor.data_ptr<float>();

    int total_output = batch_size * out_channels * out_length / blockDim.x;
    int threads = 32;
    int blocks = total_output;

    auto stream = at::cuda::getCurrentCUDAStream();
    conv_transposed_1d_kernel<<<blocks, threads, 0, stream>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch_size,
        in_channels,
        out_channels,
        in_length,
        out_length,
        kernel_size,
        stride,
        padding,
        output_padding,
        groups
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output_tensor;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Unrolled Transposed 1D convolution forward (CUDA)");
}