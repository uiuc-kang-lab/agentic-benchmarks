#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__constant__ float huber_delta;
__constant__ float quadratic_coeff;
__constant__ float linear_adj;

__global__ void smooth_l1_loss_kernel(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    int n_elements
) {
    const int elements_per_thread = 4;
    const int tid = threadIdx.x;
    const int block_offset = blockIdx.x * blockDim.x * elements_per_thread;
    const int idx_start = block_offset + tid * elements_per_thread;
    float thread_sum = 0.0f;

    for (int j = 0; j < elements_per_thread; j++) {
        int idx = idx_start + j;
        if (idx >= n_elements) break;
        float diff = predictions[idx] - targets[idx];
        float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < huber_delta) ? 
            (quadratic_coeff * diff * diff) : 
            (abs_diff - linear_adj);
    }

    __shared__ float shared_sum[256];
    shared_sum[tid] = thread_sum;
    __syncthreads();

    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_sum[tid] += shared_sum[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(output, shared_sum[0] / n_elements);
    }
}

torch::Tensor smooth_l1_loss_cuda(
    torch::Tensor predictions,
    torch::Tensor targets
) {
    TORCH_CHECK(
        predictions.sizes() == targets.sizes(),
        "Input tensors must have the same shape"
    );
    TORCH_CHECK(
        predictions.is_contiguous() && targets.is_contiguous(),
        "Input tensors must be contiguous"
    );
    TORCH_CHECK(
        predictions.device().is_cuda() && targets.device().is_cuda(),
        "Inputs must be CUDA tensors"
    );

    // Set constant memory parameters
    const float h_delta = 1.0f;
    const float h_quad = 0.5f;
    const float h_lin = 0.5f;
    hipMemcpyToSymbol(HIP_SYMBOL(huber_delta), &h_delta, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(quadratic_coeff), &h_quad, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(linear_adj), &h_lin, sizeof(float));

    int n = predictions.numel();
    auto output = torch::zeros({1}, predictions.options());

    const int block_size = 256;
    const int elements_per_thread = 4;
    // Calculate grid size to exactly cover the data with minimal oversubscription
const int total_threads_needed = (n + elements_per_thread - 1) / elements_per_thread;
const int grid_size = (total_threads_needed + block_size - 1) / block_size;

    smooth_l1_loss_kernel<<<grid_size, block_size>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &smooth_l1_loss_cuda, "Smooth L1 Loss (CUDA)");
}
