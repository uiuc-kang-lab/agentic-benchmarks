#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define THREADS 256

// Optimized kernel combining efficient memory access and atomic operations
__global__ void optimized_conv_transpose2d_kernel(
    const float * __restrict__ input,
    const float * __restrict__ weight,
    const float * __restrict__ bias,
    float * __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int input_height,
    int input_width,
    int kernel_height,
    int kernel_width,
    int output_height,
    int output_width,
    int stride_h,
    int stride_w,
    int pad_h,
    int pad_w,
    int total_sum_elems,  // = in_channels * kernel_height * kernel_width
    int blocks_per_out)   // number of blocks to partition summation for each output pixel
{
    int global_blk_idx = blockIdx.x;
    int out_pixel_idx = global_blk_idx / blocks_per_out;
    int block_in_pixel = global_blk_idx % blocks_per_out;

    int tmp = out_pixel_idx;
    int out_x = tmp % output_width; tmp /= output_width;
    int out_y = tmp % output_height; tmp /= output_height;
    int out_ch = tmp % out_channels; tmp /= out_channels;
    int batch = tmp;

    float local_sum = 0.0f;
    for (int i = block_in_pixel + threadIdx.x * blocks_per_out; i < total_sum_elems; i += blocks_per_out * blockDim.x) {
        int ic = i / (kernel_height * kernel_width);
        int rem = i % (kernel_height * kernel_width);
        int kh = rem / kernel_width;
        int kw = rem % kernel_width;

        int in_x = out_x + pad_w - kw;
        int in_y = out_y + pad_h - kh;
        if ((in_x % stride_w) == 0 && (in_y % stride_h) == 0) {
            in_x /= stride_w;
            in_y /= stride_h;
            if (in_x >= 0 && in_x < input_width && in_y >= 0 && in_y < input_height) {
                float inp_val = input[batch * in_channels * input_height * input_width +
                                        ic * input_height * input_width +
                                        in_y * input_width + in_x];
                float w = weight[ic * out_channels * kernel_height * kernel_width +
                                 out_ch * kernel_height * kernel_width +
                                 kh * kernel_width + kw];
                local_sum += inp_val * w;
            }
        }
    }

    __shared__ float smem[THREADS];
    int tid = threadIdx.x;
    smem[tid] = local_sum;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s /= 2) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        float block_sum = smem[0];
        if (block_in_pixel == 0 && bias != nullptr) {
            block_sum += bias[out_ch];
        }
        int out_idx = batch * out_channels * output_height * output_width +
                      out_ch * output_height * output_width +
                      out_y * output_width + out_x;
        atomicAdd(&output[out_idx], block_sum);
    }
}

// Host function to prepare and launch the kernel
torch::Tensor optimized_conv_transpose2d(
    torch::Tensor input,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> output_padding,
    std::vector<int64_t> dilation,
    int64_t groups) {

    int batch_size = input.size(0);
    int in_channels = input.size(1);
    int input_height = input.size(2);
    int input_width = input.size(3);
    int out_channels = weight.size(1);
    int kernel_height = weight.size(2);
    int kernel_width = weight.size(3);

    int output_height = (input_height - 1) * stride[0] - 2 * padding[0] + kernel_height + output_padding[0];
    int output_width  = (input_width - 1)  * stride[1] - 2 * padding[1] + kernel_width + output_padding[1];

    auto output = torch::zeros({batch_size, out_channels, output_height, output_width}, input.options());

    int total_sum_elems = in_channels * kernel_height * kernel_width;
    int blocks_per_out = (total_sum_elems + THREADS - 1) / THREADS;

    int num_output_pixels = batch_size * out_channels * output_height * output_width;
    int total_blocks = num_output_pixels * blocks_per_out;

    dim3 blocks(total_blocks);
    dim3 threads(THREADS);

    const float* bias_ptr = nullptr;
    if (bias.has_value() && bias.value().defined()) {
        bias_ptr = bias.value().data_ptr<float>();
    }

    optimized_conv_transpose2d_kernel<<<blocks, threads>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        out_channels,
        input_height,
        input_width,
        kernel_height,
        kernel_width,
        output_height,
        output_width,
        stride[0],
        stride[1],
        padding[0],
        padding[1],
        total_sum_elems,
        blocks_per_out
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &optimized_conv_transpose2d, "Optimized ConvTranspose2D forward (CUDA)");
}
