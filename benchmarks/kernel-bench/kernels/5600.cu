#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <limits>

// This unified kernel combines the 2D thread block mapping for spatial dimensions (for better memory coalescing)
// with compile-time unrolling for common kernel sizes (2 and 3) and a dynamic loop for other sizes. It also uses
// __ldg for read-only caching. The third grid dimension indexes the combined (batch, channel) dimension.

template <typename scalar_t, int KERNEL_SIZE>
__global__ void efficient_max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,  // used only for dynamic (KERNEL_SIZE == -1)
    const int stride,
    const int padding,
    const int dilation
) {
    // Map threads to output spatial coordinates
    int ow = blockIdx.x * blockDim.x + threadIdx.x;
    int oh = blockIdx.y * blockDim.y + threadIdx.y;
    int bc = blockIdx.z;  // combined (batch, channel)

    if (ow >= output_width || oh >= output_height) return;

    int b = bc / channels;
    int c = bc % channels;

    int input_batch_stride = channels * input_height * input_width;
    int input_channel_stride = input_height * input_width;
    int base_idx = b * input_batch_stride + c * input_channel_stride;

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    int ih_start = oh * stride - padding;
    int iw_start = ow * stride - padding;

    if constexpr (KERNEL_SIZE == 2) {
        #pragma unroll
        for (int kh = 0; kh < 2; ++kh) {
            int ih = ih_start + kh * dilation;
            if (ih < 0 || ih >= input_height) continue;
            int row_idx = base_idx + ih * input_width;
            #pragma unroll
            for (int kw = 0; kw < 2; ++kw) {
                int iw = iw_start + kw * dilation;
                if (iw < 0 || iw >= input_width) continue;
                max_val = max(max_val, __ldg(&input[row_idx + iw]));
            }
        }
    } else if constexpr (KERNEL_SIZE == 3) {
        #pragma unroll
        for (int kh = 0; kh < 3; ++kh) {
            int ih = ih_start + kh * dilation;
            if (ih < 0 || ih >= input_height) continue;
            int row_idx = base_idx + ih * input_width;
            #pragma unroll
            for (int kw = 0; kw < 3; ++kw) {
                int iw = iw_start + kw * dilation;
                if (iw < 0 || iw >= input_width) continue;
                max_val = max(max_val, __ldg(&input[row_idx + iw]));
            }
        }
    } else if constexpr (KERNEL_SIZE > 0) {
        // For other compile-time known kernel sizes
        for (int kh = 0; kh < KERNEL_SIZE; ++kh) {
            int ih = ih_start + kh * dilation;
            if (ih < 0 || ih >= input_height) continue;
            int row_idx = base_idx + ih * input_width;
            for (int kw = 0; kw < KERNEL_SIZE; ++kw) {
                int iw = iw_start + kw * dilation;
                if (iw < 0 || iw >= input_width) continue;
                max_val = max(max_val, __ldg(&input[row_idx + iw]));
            }
        }
    } else {
        // Dynamic kernel size case (KERNEL_SIZE == -1)
        for (int kh = 0; kh < kernel_size; ++kh) {
            int ih = ih_start + kh * dilation;
            if (ih < 0 || ih >= input_height) continue;
            int row_idx = base_idx + ih * input_width;
            for (int kw = 0; kw < kernel_size; ++kw) {
                int iw = iw_start + kw * dilation;
                if (iw < 0 || iw >= input_width) continue;
                max_val = max(max_val, __ldg(&input[row_idx + iw]));
            }
        }
    }

    int out_idx = b * (channels * output_height * output_width) +
                  c * (output_height * output_width) +
                  oh * output_width + ow;
    output[out_idx] = max_val;
}


// Host function to launch the kernel
torch::Tensor efficient_max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const int batch_size = input.size(0);
    const int channels = input.size(1);
    const int input_height = input.size(2);
    const int input_width = input.size(3);
    const int output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const int output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    // Use a 2D thread block for output spatial dimensions and a 3D grid (third dimension for batch*channels).
    dim3 threads(16, 16);
    dim3 blocks((output_width + threads.x - 1) / threads.x,
                (output_height + threads.y - 1) / threads.y,
                batch_size * channels);

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "efficient_max_pool2d_cuda_forward", ([&] {
        if (kernel_size == 2) {
            efficient_max_pool2d_kernel<scalar_t, 2><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size,
                channels,
                input_height,
                input_width,
                output_height,
                output_width,
                kernel_size, // not used in this branch
                stride,
                padding,
                dilation
            );
        } else if (kernel_size == 3) {
            efficient_max_pool2d_kernel<scalar_t, 3><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size,
                channels,
                input_height,
                input_width,
                output_height,
                output_width,
                kernel_size, // not used in this branch
                stride,
                padding,
                dilation
            );
        } else {
            // For dynamic or other kernel sizes, use KERNEL_SIZE = -1
            efficient_max_pool2d_kernel<scalar_t, -1><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size,
                channels,
                input_height,
                input_width,
                output_height,
                output_width,
                kernel_size,
                stride,
                padding,
                dilation
            );
        }
    }));

    return output;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &efficient_max_pool2d_cuda_forward, "Efficient Max Pool 2D forward (CUDA)");
}
