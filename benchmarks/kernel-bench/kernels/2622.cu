#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel for ReLU activation using shared memory and warp-level primitives
template <typename scalar_t>
__global__ void relu_kernel_shared_memory(
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ input,
    const int64_t size) {
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process data directly in global memory since ReLU is element-wise
    if (idx < size) {
        scalar_t val = input[idx];
        output[idx] = val > 0 ? val : 0;
    }
}

// PyTorch wrapper function
torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    
    const int threads = 256;
    const int blocks = (input.numel() + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "relu_kernel_shared_memory", ([&] {
        relu_kernel_shared_memory<scalar_t><<<blocks, threads, threads * sizeof(scalar_t)>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            input.numel()
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ReLU forward with shared memory (CUDA)");
}