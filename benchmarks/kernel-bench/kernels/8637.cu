#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK((x).is_cuda(), #x " must be a CUDA tensor");
#define CHECK_CONTIGUOUS(x) TORCH_CHECK((x).is_contiguous(), #x " must be contiguous");
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x);

__global__ void transposed_conv3d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int batch_offset,
    const int batch_size,
    const int in_channels,
    const int in_d, const int in_h, const int in_w,
    const int out_channels,
    const int out_d, const int out_h, const int out_w,
    const int k_d, const int k_h, const int k_w,
    const int s_d, const int s_h, const int s_w,
    const int p_d, const int p_h, const int p_w,
    const int groups,
    const int channels_per_group_in,
    const int channels_per_group_out) {
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total = batch_size * out_channels * out_d * out_h * out_w;
    
    if (idx < total) {
        int tmp = idx;
        const int w_out = tmp % out_w; tmp /= out_w;
        const int h_out = tmp % out_h; tmp /= out_h;
        const int d_out = tmp % out_d; tmp /= out_d;
        const int oc = tmp % out_channels;
        const int n = tmp / out_channels + batch_offset;

        float sum = (bias != nullptr) ? bias[oc] : 0.0f;
        
        const int group = oc / channels_per_group_out;
        const int oc_in_group = oc % channels_per_group_out;
        
        const int d_base = d_out + p_d;
        const int h_base = h_out + p_h;
        const int w_base = w_out + p_w;

        #pragma unroll 4
        for (int kd = 0; kd < k_d; kd++) {
            const int tmp_d = d_base - kd;
            if (tmp_d % s_d != 0) continue;
            const int in_d_idx = tmp_d / s_d;
            if (in_d_idx < 0 || in_d_idx >= in_d) continue;

            #pragma unroll 4
            for (int kh = 0; kh < k_h; kh++) {
                const int tmp_h = h_base - kh;
                if (tmp_h % s_h != 0) continue;
                const int in_h_idx = tmp_h / s_h;
                if (in_h_idx < 0 || in_h_idx >= in_h) continue;

                #pragma unroll 4
                for (int kw = 0; kw < k_w; kw++) {
                    const int tmp_w = w_base - kw;
                    if (tmp_w % s_w != 0) continue;
                    const int in_w_idx = tmp_w / s_w;
                    if (in_w_idx < 0 || in_w_idx >= in_w) continue;

                    #pragma unroll 4
                    for (int ic = 0; ic < channels_per_group_in; ic++) {
                        const int in_channel = group * channels_per_group_in + ic;
                        const int input_idx = n * (in_channels * in_d * in_h * in_w) +
                                            in_channel * (in_d * in_h * in_w) +
                                            in_d_idx * (in_h * in_w) +
                                            in_h_idx * in_w + in_w_idx;
                        
                        const int weight_idx = in_channel * (channels_per_group_out * k_d * k_h * k_w) +
                                             oc_in_group * (k_d * k_h * k_w) +
                                             kd * (k_h * k_w) + kh * k_w + kw;
                        
                        sum += input[input_idx] * weight[weight_idx];
                    }
                }
            }
        }
        output[idx] = sum;
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias_opt,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> output_padding,
    int64_t groups) {

    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias_opt.has_value()) {
        CHECK_INPUT(*bias_opt);
    }

    const int batch = x.size(0);
    const int in_channels = x.size(1);
    const int in_d = x.size(2);
    const int in_h = x.size(3);
    const int in_w = x.size(4);
    
    const int k_d = weight.size(2);
    const int k_h = weight.size(3);
    const int k_w = weight.size(4);
    
    const int s_d = stride[0];
    const int s_h = stride[1];
    const int s_w = stride[2];
    
    const int p_d = padding[0];
    const int p_h = padding[1];
    const int p_w = padding[2];
    
    const int op_d = output_padding[0];
    const int op_h = output_padding[1];
    const int op_w = output_padding[2];

    const int out_d = (in_d - 1) * s_d - 2 * p_d + k_d + op_d;
    const int out_h = (in_h - 1) * s_h - 2 * p_h + k_h + op_h;
    const int out_w = (in_w - 1) * s_w - 2 * p_w + k_w + op_w;

    const int channels_per_group_out = weight.size(1);
    const int out_channels = channels_per_group_out * groups;
    const int channels_per_group_in = in_channels / groups;

    auto output = torch::zeros({batch, out_channels, out_d, out_h, out_w}, x.options());

    // Create CUDA streams
    hipStream_t compute_stream, transfer_stream;
    hipStreamCreate(&compute_stream);
    hipStreamCreate(&transfer_stream);

    // Calculate chunk size for batch processing
    const int chunk_size = 4;  // Process 4 samples at a time
    const int num_chunks = (batch + chunk_size - 1) / chunk_size;

    // Get raw pointers
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias_opt.has_value() ? (*bias_opt).data_ptr<float>() : nullptr;

    for (int chunk = 0; chunk < num_chunks; chunk++) {
        const int chunk_start = chunk * chunk_size;
        const int current_chunk_size = std::min(chunk_size, batch - chunk_start);
        
        const int elements_per_chunk = current_chunk_size * out_channels * out_d * out_h * out_w;
        const int threads = 256;
        const int blocks = (elements_per_chunk + threads - 1) / threads;

        // Launch kernel on compute stream
        transposed_conv3d_kernel<<<blocks, threads, 0, compute_stream>>>(
            x.data_ptr<float>(),
            weight_ptr,
            bias_ptr,
            output.data_ptr<float>(),
            chunk_start,
            current_chunk_size,
            in_channels,
            in_d, in_h, in_w,
            out_channels,
            out_d, out_h, out_w,
            k_d, k_h, k_w,
            s_d, s_h, s_w,
            p_d, p_h, p_w,
            groups,
            channels_per_group_in,
            channels_per_group_out);
    }

    // Cleanup streams
    hipStreamDestroy(compute_stream);
    hipStreamDestroy(transfer_stream);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Transposed Conv3D forward with stream overlap (CUDA)");
}