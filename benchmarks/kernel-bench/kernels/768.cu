#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32
#define TILE_SIZE 16

__global__ void matmul_warp_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    const unsigned int FULL_MASK = 0xffffffff;
    
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    float sum = 0.0f;
    
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    // Compute matrix multiplication
    for (int i = 0; i < K; i += TILE_SIZE) {
        // Load data into shared memory
        if (row < M && (i + threadIdx.x) < K) {
            As[threadIdx.y][threadIdx.x] = A[row * K + i + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        if ((i + threadIdx.y) < K && col < N) {
            Bs[threadIdx.y][threadIdx.x] = B[(i + threadIdx.y) * N + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute partial dot product for this tile
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            sum = __fmaf_rn(As[threadIdx.y][k], Bs[k][threadIdx.x], sum);
        }
        
        __syncthreads();
    }
    
    // Write result
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.is_contiguous(), "A must be contiguous");
    TORCH_CHECK(B.is_contiguous(), "B must be contiguous");
    
    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);
    
    torch::Tensor C = torch::zeros({M, N}, A.options());
    
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE,
                   (M + TILE_SIZE - 1) / TILE_SIZE);
    
    matmul_warp_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        M, N, K
    );
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Warp-optimized Matrix Multiplication (CUDA)");
}