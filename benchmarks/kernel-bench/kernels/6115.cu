#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void avg_pool2d_forward_kernel_unroll(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int N,
    int C,
    int H,
    int W,
    int outH,
    int outW,
    int kernel_size,
    int stride,
    int padding
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = N * C * outH * outW;
    if (index >= total) {
        return;
    }

    int w_out = index % outW;
    int h_out = (index / outW) % outH;
    int c = (index / (outW * outH)) % C;
    int n = index / (outW * outH * C);

    int h_start = h_out * stride - padding;
    int w_start = w_out * stride - padding;

    scalar_t sum_val = scalar_t(0);
    #pragma unroll
    for (int i = 0; i < kernel_size; i++) {
        #pragma unroll
        for (int j = 0; j < kernel_size; j++) {
            int h_in = h_start + i;
            int w_in = w_start + j;
            if (h_in >= 0 && h_in < H && w_in >= 0 && w_in < W) {
                sum_val += input[((n * C + c) * H + h_in) * W + w_in];
            }
        }
    }
    output[index] = sum_val / static_cast<scalar_t>(kernel_size * kernel_size);
}

torch::Tensor avg_pool2d_forward_unroll(
    torch::Tensor x,
    int kernel_size,
    int stride,
    int padding
) {
    TORCH_CHECK(x.dim() == 4, "Input must be a 4D tensor.");
    auto N = x.size(0);
    auto C = x.size(1);
    auto H = x.size(2);
    auto W = x.size(3);

    int outH = (H + 2 * padding - kernel_size) / stride + 1;
    int outW = (W + 2 * padding - kernel_size) / stride + 1;

    auto x_cont = x.contiguous();
    auto options = x.options();
    auto out = torch::empty({N, C, outH, outW}, options);

    const int threads = 256;
    const int blocks = (N * C * outH * outW + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "avg_pool2d_forward_kernel_unroll", ([&] {
        const scalar_t* input_data = x_cont.data_ptr<scalar_t>();
        scalar_t* output_data = out.data_ptr<scalar_t>();

        avg_pool2d_forward_kernel_unroll<<<blocks, threads>>>(
            input_data,
            output_data,
            N, C, H, W,
            outH, outW,
            kernel_size, stride, padding
        );
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err));

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &avg_pool2d_forward_unroll, "2D Average Pooling forward with loop unrolling (CUDA)");
}