#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <float.h>

__global__ void argmax_kernel_coop_sync_opt(
    const float* __restrict__ x,
    int64_t* __restrict__ indices,
    const int outerSize,
    const int dimSize,
    const int innerSize) {

    int slice = blockIdx.x;
    if (slice >= outerSize * innerSize) return;

    int outer_idx = slice / innerSize;
    int inner_idx = slice % innerSize;
    int base_offset = outer_idx * (dimSize * innerSize) + inner_idx;

    float local_max = -FLT_MAX;
    int local_argmax = 0;

    for (int d = threadIdx.x; d < dimSize; d += blockDim.x) {
        float curr_val = x[base_offset + d * innerSize];
        if (curr_val > local_max) {
            local_max = curr_val;
            local_argmax = d;
        }
    }

    extern __shared__ char shared_mem[];
    float* s_max = reinterpret_cast<float*>(shared_mem);
    int* s_idx = reinterpret_cast<int*>(s_max + blockDim.x);

    s_max[threadIdx.x] = local_max;
    s_idx[threadIdx.x] = local_argmax;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (threadIdx.x < s) {
            if (s_max[threadIdx.x + s] > s_max[threadIdx.x]) {
                s_max[threadIdx.x] = s_max[threadIdx.x + s];
                s_idx[threadIdx.x] = s_idx[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    if (threadIdx.x < 32) {
        if (s_max[threadIdx.x + 32] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 32];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 32];
        }
        if (s_max[threadIdx.x + 16] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 16];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 16];
        }
        if (s_max[threadIdx.x + 8] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 8];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 8];
        }
        if (s_max[threadIdx.x + 4] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 4];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 4];
        }
        if (s_max[threadIdx.x + 2] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 2];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 2];
        }
        if (s_max[threadIdx.x + 1] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 1];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 1];
        }
    }

    if (threadIdx.x == 0) {
        indices[slice] = s_idx[0];
    }
}

torch::Tensor argmax_forward_cuda(const torch::Tensor& x, const int64_t dim) {
    TORCH_CHECK(x.scalar_type() == at::kFloat, "Only float32 is supported.");
    auto x_contig = x.contiguous();

    auto sizes = x_contig.sizes();
    int ndim = x_contig.dim();
    TORCH_CHECK(dim >= 0 && dim < ndim, "Invalid dim for argmax.");

    int outerSize = 1;
    for (int d = 0; d < dim; d++) {
        outerSize *= sizes[d];
    }
    int dimSize = sizes[dim];
    int innerSize = 1;
    for (int d = dim + 1; d < ndim; d++) {
        innerSize *= sizes[d];
    }

    std::vector<int64_t> out_sizes;
    for (int d = 0; d < ndim; d++) {
        if (d == dim) continue;
        out_sizes.push_back(sizes[d]);
    }
    auto options = torch::TensorOptions().device(x.device()).dtype(torch::kLong);
    auto indices = torch::empty(out_sizes, options);

    int slices = outerSize * innerSize;
    const int threads = 128;
    int blocks = slices;
    int sharedMemSize = threads * (sizeof(float) + sizeof(int));

    argmax_kernel_coop_sync_opt<<<blocks, threads, sharedMemSize>>>(
        x_contig.data_ptr<float>(),
        indices.data_ptr<int64_t>(),
        outerSize,
        dimSize,
        innerSize
    );

    return indices;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &argmax_forward_cuda, "ArgMax CUDA forward with optimized synchronization");
}