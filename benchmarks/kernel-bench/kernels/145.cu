#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
#define TILE_SIZE 16

// Define CHECK_INPUT macro for tensor validation
#define CHECK_INPUT(x) \
    TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor"); \
    TORCH_CHECK(x.is_contiguous(), #x " must be contiguous"); \
    TORCH_CHECK(x.scalar_type() == torch::kFloat, #x " must be a float tensor");

__global__ void matrix_multiply_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    
    float sum = 0.0f;
    
    for (int t = 0; t < (K - 1) / TILE_SIZE + 1; ++t) {
        if (row < M && t * TILE_SIZE + tx < K)
            As[ty][tx] = A[row * K + t * TILE_SIZE + tx];
        else
            As[ty][tx] = 0.0f;
            
        if (t * TILE_SIZE + ty < K && col < N)
            Bs[ty][tx] = B[(t * TILE_SIZE + ty) * N + col];
        else
            Bs[ty][tx] = 0.0f;
            
        __syncthreads();
        
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[ty][k] * Bs[k][tx];
        }
        __syncthreads();
    }
    
    if (row < M && col < N)
        C[row * N + col] = sum;
}

void matrix_multiply_cuda(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE,
                   (M + TILE_SIZE - 1) / TILE_SIZE);

    matrix_multiply_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        M, N, K
    );
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());
    matrix_multiply_cuda(A, B, C);
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication (CUDA)");
}