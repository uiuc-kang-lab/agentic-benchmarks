#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define TILE_WIDTH 32

template <typename scalar_t>
__global__ void max_pool2d_kernel_shared(
    const scalar_t* __restrict__ input,
    scalar_t* output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    extern __shared__ scalar_t tile[];

    const int tid = threadIdx.x;
    const int batch_idx = blockIdx.z;
    const int channel_idx = blockIdx.y;
    const int row = blockIdx.x / ((output_width + TILE_WIDTH - 1) / TILE_WIDTH) * TILE_WIDTH + threadIdx.x / TILE_WIDTH;
    const int col = (blockIdx.x % ((output_width + TILE_WIDTH - 1) / TILE_WIDTH)) * TILE_WIDTH + threadIdx.x % TILE_WIDTH;

    if (row < output_height && col < output_width) {
        scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

        for (int kh = 0; kh < kernel_size; kh++) {
            const int ih = row * stride - padding + kh * dilation;

            if (ih >= 0 && ih < input_height) {
                for (int kw = 0; kw < kernel_size; kw++) {
                    const int iw = col * stride - padding + kw * dilation;

                    if (iw >= 0 && iw < input_width) {
                        const int input_idx = batch_idx * (channels * input_height * input_width) +
                                            channel_idx * (input_height * input_width) +
                                            ih * input_width +
                                            iw;
                        tile[tid] = __ldg(&input[input_idx]);
                        max_val = max(max_val, tile[tid]);
                    }
                }
            }
        }

        const int output_idx = batch_idx * (channels * output_height * output_width) +
                              channel_idx * (output_height * output_width) +
                              row * output_width +
                              col;
        output[output_idx] = max_val;
    }
}

torch::Tensor max_pool2d_cuda_forward_shared(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const dim3 threads(TILE_WIDTH * TILE_WIDTH);
    const dim3 blocks(
        ((output_height + TILE_WIDTH - 1) / TILE_WIDTH) * ((output_width + TILE_WIDTH - 1) / TILE_WIDTH),
        channels,
        batch_size
    );

    const size_t shared_memory_size = TILE_WIDTH * TILE_WIDTH * sizeof(scalar_t);

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool2d_cuda_forward_shared", ([&] {
        max_pool2d_kernel_shared<scalar_t><<<blocks, threads, shared_memory_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward_shared, "Max Pool 2D forward with shared memory (CUDA)");
}