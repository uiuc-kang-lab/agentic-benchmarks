#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__device__ float compute_softsign(float value) {
    return value / (1.0f + fabsf(value));
}

__global__ void softsign_kernel_modular(const float* __restrict__ x, float* __restrict__ out, int num_elements) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    // Grid-stride loop with modular device function call
    for (int idx = tid; idx < num_elements; idx += stride) {
        out[idx] = compute_softsign(x[idx]);
    }
}

torch::Tensor forward(torch::Tensor x) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int num_elements = x.numel();
    int threads = 1024;
    int blocks = (num_elements + threads - 1) / threads;

    softsign_kernel_modular<<<blocks, threads>>>(
        x.data_ptr<float>(), out.data_ptr<float>(), num_elements
    );

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Softsign activation with modular device functions (CUDA)");
}