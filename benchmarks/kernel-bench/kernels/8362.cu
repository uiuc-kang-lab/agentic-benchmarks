#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/util/Optional.h>

namespace py = pybind11;

__global__ void conv1d_forward_kernel(
    const float* __restrict__ x,
    const float* __restrict__ w,
    const float* __restrict__ bias,
    float* __restrict__ y,
    const int N,
    const int C_in,
    const int L_in,
    const int C_out,
    const int K,
    const int stride,
    const int padding,
    const int dilation,
    const int groups,
    const int L_out,
    const int group_size_in,
    const int group_size_out
) {
    extern __shared__ float shared_mem[];
    float* shared_x = shared_mem;
    float* shared_w = shared_mem + blockDim.x + 2 * padding;

    const int tid = threadIdx.x;
    const int out_pos = blockIdx.x * blockDim.x + tid;
    const int out_ch = blockIdx.y;
    const int n = blockIdx.z;
    
    if (n >= N || out_ch >= C_out) return;

    const int group_idx = out_ch / group_size_out;
    float val = 0.0f;

    // Load input data into shared memory
    if (out_pos < L_out) {
        for (int local_in_ch = 0; local_in_ch < group_size_in; ++local_in_ch) {
            const int in_ch = group_idx * group_size_in + local_in_ch;
            
            // Load input window including padding
            for (int i = tid; i < blockDim.x + 2 * padding; i += blockDim.x) {
                int in_pos = blockIdx.x * blockDim.x + i - padding;
                shared_x[i] = (in_pos >= 0 && in_pos < L_in) ? 
                    x[n * C_in * L_in + in_ch * L_in + in_pos] : 0.0f;
            }
            
            // Load weights into shared memory
            for (int k = tid; k < K; k += blockDim.x) {
                shared_w[k] = w[out_ch * (group_size_in * K) + local_in_ch * K + k];
            }
            
            __syncthreads();  // Ensure shared memory is loaded

            // Compute convolution
            #pragma unroll 4
            for (int k = 0; k < K; ++k) {
                int relative_pos = tid + k * dilation;
                val += shared_x[relative_pos + padding] * shared_w[k];
            }
            
            __syncthreads();  // Ensure shared memory can be reused for next channel
        }

        if (bias) {
            val += bias[out_ch];
        }

        if (out_pos < L_out) {
            y[n * C_out * L_out + out_ch * L_out + out_pos] = val;
        }
    }
}

at::Tensor conv1d_forward_impl(
    const at::Tensor& x,
    const at::Tensor& weight,
    c10::optional<at::Tensor> bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups
) {
    TORCH_CHECK(x.is_cuda(), "x must be CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "weight must be CUDA tensor");

    const int N = x.size(0);
    const int C_in = x.size(1);
    const int L_in = x.size(2);
    const int C_out = weight.size(0);
    const int K = weight.size(2);

    const int L_out = (L_in + 2 * padding - dilation * (K - 1) - 1) / stride + 1;
    TORCH_CHECK(L_out > 0, "Invalid output length");

    auto y = torch::empty({N, C_out, L_out}, x.options());
    const float* bias_ptr = bias_opt.has_value() ? bias_opt->data_ptr<float>() : nullptr;

    const int group_size_in = C_in / groups;
    const int group_size_out = C_out / groups;

    const int block_size = 128;
    dim3 grid(
        (L_out + block_size - 1) / block_size,
        C_out,
        N
    );

    const size_t shared_mem_size = (block_size + 2 * padding + K) * sizeof(float);

    conv1d_forward_kernel<<<grid, block_size, shared_mem_size>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        y.data_ptr<float>(),
        N, C_in, L_in, C_out, K,
        stride, padding, dilation, groups, L_out,
        group_size_in, group_size_out
    );

    TORCH_CHECK(hipGetLastError() == hipSuccess, "Kernel launch failed");
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward",
        [](at::Tensor x, at::Tensor weight, py::object bias,
           int64_t stride, int64_t padding, int64_t dilation, int64_t groups) {
            return conv1d_forward_impl(x, weight,
                bias.is_none() ? c10::nullopt : c10::optional<at::Tensor>(bias.cast<at::Tensor>()),
                stride, padding, dilation, groups);
        }, "Shared memory optimized 1D Conv"
    );
}