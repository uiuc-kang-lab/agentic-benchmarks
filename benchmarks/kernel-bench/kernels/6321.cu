#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Modular device function for performing sum reduction over a dimension
template <typename scalar_t>
__device__ inline scalar_t compute_sum(const scalar_t* input, int64_t base_idx, int64_t reduce_size, int64_t inner_size) {
    scalar_t sum = 0;
    #pragma unroll
    for (int64_t i = 0; i < reduce_size; i++) {
        sum += input[base_idx + i * inner_size];
    }
    return sum;
}

// Kernel that utilizes the modular device function
template <typename scalar_t>
__global__ void modular_sum_reduce_kernel(
    const scalar_t* input,
    scalar_t* output,
    int64_t reduce_size,
    int64_t outer_size,
    int64_t inner_size) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= outer_size * inner_size) return;
    
    int outer_idx = idx / inner_size;
    int inner_idx = idx % inner_size;
    int64_t base_idx = outer_idx * reduce_size * inner_size + inner_idx;
    
    // Use the device function to compute the sum reduction
    scalar_t sum = compute_sum(input, base_idx, reduce_size, inner_size);
    output[outer_idx * inner_size + inner_idx] = sum;
}

// CUDA wrapper function
torch::Tensor sum_reduce_cuda(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();
    
    auto sizes = input.sizes().vec();
    int64_t reduce_size = sizes[dim];
    
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }
    
    sizes[dim] = 1;
    auto output = torch::empty(sizes, input.options());
    
    const int threads = 512;
    const int blocks = (outer_size * inner_size + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sum_reduce_cuda", ([&] {
        modular_sum_reduce_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            reduce_size,
            outer_size,
            inner_size
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &sum_reduce_cuda, "Modular sum reduction forward (CUDA)");
}
