#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHUNK_SIZE 1024
#define NUM_STREAMS 4

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   int N,
                                   int chunk_start) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    row += chunk_start;  // Offset by chunk start
    
    if (row < N && col < N) {
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            float sum = 0.f;
            #pragma unroll
            for (int k = col; k <= row; ++k) {
                sum += A[row * N + k] * B[k * N + col];
            }
            C[row * N + col] = sum;
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D tensors");
    TORCH_CHECK(A.size(0) == A.size(1) && B.size(0) == B.size(1), "Matrices must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "Matrices must have same dimensions");

    const int N = A.size(0);
    auto C = torch::empty_like(A);

    // Create CUDA streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    // Configure kernel dimensions
    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);

    // Process matrix in chunks using different streams
    for (int chunk_start = 0; chunk_start < N; chunk_start += CHUNK_SIZE) {
        int chunk_size = min(CHUNK_SIZE, N - chunk_start);
        dim3 numBlocks((N + threads - 1) / threads,
                      (chunk_size + threads - 1) / threads);

        int stream_idx = (chunk_start / CHUNK_SIZE) % NUM_STREAMS;
        
        triangular_mm_kernel<<<numBlocks, threadsPerBlock, 0, streams[stream_idx]>>>(
            A.data_ptr<float>(),
            B.data_ptr<float>(),
            C.data_ptr<float>(),
            N,
            chunk_start
        );
    }

    // Synchronize all streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}