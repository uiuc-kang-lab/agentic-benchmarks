#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
#define WARP_SIZE 32

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   int N) {
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    
    int bx = blockIdx.x * BLOCK_SIZE;
    int by = blockIdx.y * BLOCK_SIZE;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by + ty;
    int col = bx + tx;
    
    float sum = 0.0f;
    
    // Only compute for lower triangular portion
    if (row >= col && row < N && col < N) {
        // Loop over blocks to compute partial sums
        for (int m = col; m <= row; m += BLOCK_SIZE) {
            // Load data into shared memory
            if (m + tx <= row && by + ty < N) {
                As[ty][tx] = A[row * N + (m + tx)];
            } else {
                As[ty][tx] = 0.0f;
            }
            
            if (m + ty <= N-1 && bx + tx < N) {
                Bs[ty][tx] = B[(m + ty) * N + col];
            } else {
                Bs[ty][tx] = 0.0f;
            }
            
            __syncthreads();
            
            // Compute partial sum for this tile
            #pragma unroll
            for (int k = 0; k < BLOCK_SIZE; k++) {
                if (m + k <= row) {
                    sum += As[ty][k] * Bs[k][tx];
                }
            }
            
            __syncthreads();
        }
        
        // Warp-level reduction for final sum
        #pragma unroll
        for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }
        
        // Write result
        if (row < N && col < N) {
            if (row >= col) {
                C[row * N + col] = sum;
            } else {
                C[row * N + col] = 0.0f;
            }
        }
    }
}

// C++ interface exposed to PyTorch
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
                   (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}