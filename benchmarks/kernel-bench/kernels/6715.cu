#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define TILE_SIZE 256
#define WARP_SIZE 32

__global__ void unrolled_prod_reduce_kernel(const float* __restrict__ input,
                                             float* __restrict__ output,
                                             const int dim_size,
                                             const int stride) {
    __shared__ float shared_data[TILE_SIZE];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int lane_id = tid % WARP_SIZE;
    const int warp_id = tid / WARP_SIZE;

    // Initialize partial product
    float thread_prod = 1.0f;

    // Process input in tiles, unrolling the loop manually
    for (int tile_start = 0; tile_start < dim_size; tile_start += TILE_SIZE) {
        shared_data[tid] = 1.0f;
        __syncthreads();

        const int tile_end = min(tile_start + TILE_SIZE, dim_size);
        #pragma unroll 4
        for (int i = tile_start + tid; i < tile_end; i += blockDim.x) {
            shared_data[tid] *= input[bid + i * stride];
        }
        __syncthreads();

        if (tid < WARP_SIZE) {
            float warp_prod = 1.0f;
            #pragma unroll 4
            for (int i = tid; i < TILE_SIZE; i += WARP_SIZE) {
                warp_prod *= shared_data[i];
            }

            // Warp-level reduction using shuffle
            #pragma unroll
            for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1) {
                warp_prod *= __shfl_down_sync(0xffffffff, warp_prod, offset);
            }

            // First thread writes its warp's product to thread product
            if (lane_id == 0) {
                thread_prod *= warp_prod;
            }
        }
    }

    // Final warp-level reduction
    if (warp_id == 0 && lane_id == 0) {
        float final_prod = thread_prod;
        for (int offset = NUM_WARPS / 2; offset > 0; offset >>= 1) {
            final_prod *= __shfl_down_sync(0xffffffff, final_prod, offset);
        }
        if (lane_id == 0) {
            output[bid] = final_prod;
        }
    }
}

torch::Tensor forward(torch::Tensor x, int dim) {
    CHECK_INPUT(x);

    auto sizes = x.sizes().vec();
    int dim_size = sizes[dim];
    sizes.erase(sizes.begin() + dim);
    torch::Tensor output = torch::empty(sizes, x.options());

    int num_elements = output.numel();
    int stride = x.stride(dim);

    const float* input_ptr = x.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();

    // Launch configuration
    int threads = TILE_SIZE;
    int blocks = num_elements;

    unrolled_prod_reduce_kernel<<<blocks, threads>>>(input_ptr, output_ptr, dim_size, stride);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Unrolled loop product reduction (CUDA)");
}