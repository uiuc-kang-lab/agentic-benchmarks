#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel using a stride loop for correct boundary handling
// Each thread processes multiple elements in the input tensor

template <typename scalar_t>
__global__ void stride_loop_relu_kernel(
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ input,
    const int64_t size) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Process elements with a grid-stride loop
    for (int i = tid; i < size; i += stride) {
        scalar_t val = input[i];
        output[i] = val > static_cast<scalar_t>(0) ? val : static_cast<scalar_t>(0);
    }
}

// PyTorch wrapper function
torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int threads = 256;
    const int blocks = (input.numel() + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "stride_loop_relu_kernel", ([&] {
        stride_loop_relu_kernel<scalar_t><<<blocks, threads>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            input.numel()
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Stride Loop ReLU forward (CUDA)");
}
