#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>

namespace py = pybind11;

#define TILE_SIZE 16

// Depthwise convolution kernel with branchless bounds checking to minimize warp divergence.
template <typename scalar_t>
__global__ void branchless_depthwise_conv2d_kernel(
    const scalar_t* __restrict__ input,   // [batch, channels, in_h, in_w]
    const scalar_t* __restrict__ weight,  // [channels, 1, k, k]
    const scalar_t* __restrict__ bias,    // [channels] or nullptr
    scalar_t* __restrict__ output,        // [batch, channels, out_h, out_w]
    int batch,
    int channels,
    int in_h, int in_w,
    int out_h, int out_w,
    int k,
    int stride,
    int padding,
    int dilation) {
  
  // Each thread is responsible for one output pixel in a given (batch, channel)
  int linear_idx = blockIdx.z;
  int n = linear_idx / channels;
  int c = linear_idx % channels;

  int ow = blockIdx.x * blockDim.x + threadIdx.x;
  int oh = blockIdx.y * blockDim.y + threadIdx.y;

  // Only compute if within output bounds
  if (oh < out_h && ow < out_w) {
    scalar_t sum = (bias != nullptr) ? bias[c] : static_cast<scalar_t>(0);
    
    // Loop over the kernel window
    for (int i = 0; i < k; i++) {
      // Compute the input row index for the kernel element
      int ih = oh * stride - padding + i * dilation;
      
      // Compute branchless indicator for row validity
      // Using unsigned comparison to avoid branching
      int valid_ih = ((unsigned)ih < (unsigned)in_h);
      // If not valid, use 0 as safe index (reads from the first row, which is safe because it exists)
      int safe_ih = valid_ih ? ih : 0;
      
      for (int j = 0; j < k; j++) {
        int iw = ow * stride - padding + j * dilation;
        int valid_iw = ((unsigned)iw < (unsigned)in_w);
        int safe_iw = valid_iw ? iw : 0;
        
        // Overall validity (1 if both valid, 0 otherwise)
        int valid = valid_ih * valid_iw;

        // Compute linear indices with safe indices for out-of-bound accesses
        int input_idx = n * channels * in_h * in_w + c * in_h * in_w + safe_ih * in_w + safe_iw;
        int weight_idx = c * k * k + i * k + j;

        // Multiply by valid mask to nullify contributions when out-of-bound
        sum += valid * input[input_idx] * weight[weight_idx];
      }
    }
    
    int output_idx = n * channels * out_h * out_w + c * out_h * out_w + oh * out_w + ow;
    output[output_idx] = sum;
  }
}

// Pointwise convolution kernel (1x1 conv) using conventional uniform control flow
template <typename scalar_t>
__global__ void branchless_pointwise_conv2d_kernel(
    const scalar_t* __restrict__ input,   // [batch, in_channels, h, w]
    const scalar_t* __restrict__ weight,  // [out_channels, in_channels]
    const scalar_t* __restrict__ bias,    // [out_channels] or nullptr
    scalar_t* __restrict__ output,        // [batch, out_channels, h, w]
    int batch,
    int in_channels,
    int out_channels,
    int h, int w) {

  int linear_idx = blockIdx.z;
  int n = linear_idx / out_channels;
  int oc = linear_idx % out_channels;

  int ow = blockIdx.x * blockDim.x + threadIdx.x;
  int oh = blockIdx.y * blockDim.y + threadIdx.y;

  if (oh < h && ow < w) {
    scalar_t sum = (bias != nullptr) ? bias[oc] : static_cast<scalar_t>(0);
    
    for (int ic = 0; ic < in_channels; ic++) {
      int input_idx = n * in_channels * h * w + ic * h * w + oh * w + ow;
      int weight_idx = oc * in_channels + ic;
      sum += input[input_idx] * weight[weight_idx];
    }

    int output_idx = n * out_channels * h * w + oc * h * w + oh * w + ow;
    output[output_idx] = sum;
  }
}

// Core CUDA forward function
torch::Tensor forward_cuda(
    const torch::Tensor& x,
    const torch::Tensor& depthwise_weight,
    const torch::Tensor& pointwise_weight,
    const torch::Tensor& depthwise_bias,
    const torch::Tensor& pointwise_bias,
    int stride,
    int padding,
    int dilation) {

  TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
  TORCH_CHECK(depthwise_weight.is_cuda(), "Depthwise weight must be a CUDA tensor");
  TORCH_CHECK(pointwise_weight.is_cuda(), "Pointwise weight must be a CUDA tensor");
  if (depthwise_bias.defined() && depthwise_bias.numel() > 0)
    TORCH_CHECK(depthwise_bias.is_cuda(), "Depthwise bias must be a CUDA tensor if provided");
  if (pointwise_bias.defined() && pointwise_bias.numel() > 0)
    TORCH_CHECK(pointwise_bias.is_cuda(), "Pointwise bias must be a CUDA tensor if provided");

  int batch = x.size(0);
  int channels = x.size(1);
  int in_h = x.size(2);
  int in_w = x.size(3);

  int k = depthwise_weight.size(2);
  int out_h = (in_h + 2 * padding - dilation * (k - 1) - 1) / stride + 1;
  int out_w = (in_w + 2 * padding - dilation * (k - 1) - 1) / stride + 1;

  auto depthwise_output = torch::empty({batch, channels, out_h, out_w}, x.options());

  dim3 block(TILE_SIZE, TILE_SIZE);
  dim3 grid((out_w + TILE_SIZE - 1) / TILE_SIZE,
            (out_h + TILE_SIZE - 1) / TILE_SIZE,
            batch * channels);

  const void* depthwise_bias_ptr = (depthwise_bias.defined() && depthwise_bias.numel() > 0) ? depthwise_bias.data_ptr() : nullptr;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "branchless_depthwise_conv2d_cuda", ([&] {
    branchless_depthwise_conv2d_kernel<scalar_t><<<grid, block>>>(
        x.data_ptr<scalar_t>(),
        depthwise_weight.data_ptr<scalar_t>(),
        reinterpret_cast<const scalar_t*>(depthwise_bias_ptr),
        depthwise_output.data_ptr<scalar_t>(),
        batch,
        channels,
        in_h, in_w,
        out_h, out_w,
        k,
        stride,
        padding,
        dilation);
  }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Depthwise kernel launch error: %s\n", hipGetErrorString(err));
  }

  int out_channels = pointwise_weight.size(0);
  auto output = torch::empty({batch, out_channels, out_h, out_w}, x.options());

  dim3 block_pw(TILE_SIZE, TILE_SIZE);
  dim3 grid_pw((out_w + TILE_SIZE - 1) / TILE_SIZE,
               (out_h + TILE_SIZE - 1) / TILE_SIZE,
               batch * out_channels);

  const void* pointwise_bias_ptr = (pointwise_bias.defined() && pointwise_bias.numel() > 0) ? pointwise_bias.data_ptr() : nullptr;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "branchless_pointwise_conv2d_cuda", ([&] {
    branchless_pointwise_conv2d_kernel<scalar_t><<<grid_pw, block_pw>>>(
        depthwise_output.data_ptr<scalar_t>(),
        pointwise_weight.data_ptr<scalar_t>(),
        reinterpret_cast<const scalar_t*>(pointwise_bias_ptr),
        output.data_ptr<scalar_t>(),
        batch,
        channels,
        out_channels,
        out_h, out_w);
  }));

  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Pointwise kernel launch error: %s\n", hipGetErrorString(err));
  }

  return output;
}

// Helper: converts a py::object to an at::Tensor (supports tensors or objects with a 'data' attribute).
at::Tensor toTensor(const py::object& obj) {
  if (obj.is_none()) {
    return at::Tensor();
  }
  try {
    return obj.cast<at::Tensor>();
  } catch (const py::cast_error& e) {
    if (py::hasattr(obj, "data")) {
      return obj.attr("data").cast<at::Tensor>();
    }
    throw std::runtime_error("Expected a torch Tensor or Parameter.");
  }
}

// Wrapper function exposed to Python
at::Tensor forward_wrapper(py::object x_obj,
                           py::object depthwise_weight_obj,
                           py::object pointwise_weight_obj,
                           py::object depthwise_bias_obj,
                           py::object pointwise_bias_obj,
                           int stride,
                           int padding,
                           int dilation) {
  auto x = toTensor(x_obj);
  auto depthwise_weight = toTensor(depthwise_weight_obj);
  auto pointwise_weight = toTensor(pointwise_weight_obj);
  auto depthwise_bias = toTensor(depthwise_bias_obj);
  auto pointwise_bias = toTensor(pointwise_bias_obj);
  
  return forward_cuda(x, depthwise_weight, pointwise_weight,
                      depthwise_bias, pointwise_bias,
                      stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward_wrapper, "CUDA branchless depthwise separable convolution forward");
}
