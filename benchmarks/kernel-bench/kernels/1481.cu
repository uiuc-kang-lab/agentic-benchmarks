#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Use a block size of 32 (matching warp and occupancy targets on NVIDIA H100)
#define BLOCK_SIZE 32

// Optimized CUDA kernel with manual inner loop unrolling
__global__ void matmul_unroll_kernel(const float* __restrict__ A,
                                       const float* __restrict__ B,
                                       float* __restrict__ C,
                                       int N) {
    __shared__ float s_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float s_B[BLOCK_SIZE][BLOCK_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * BLOCK_SIZE + ty;
    int col = blockIdx.x * BLOCK_SIZE + tx;

    float value = 0.0f;
    int tileCount = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    for (int t = 0; t < tileCount; t++) {
        // Load tile of A into shared memory
        if (row < N && (t * BLOCK_SIZE + tx) < N)
            s_A[ty][tx] = A[row * N + t * BLOCK_SIZE + tx];
        else
            s_A[ty][tx] = 0.0f;

        // Load tile of B into shared memory
        if (col < N && (t * BLOCK_SIZE + ty) < N)
            s_B[ty][tx] = B[(t * BLOCK_SIZE + ty) * N + col];
        else
            s_B[ty][tx] = 0.0f;

        __syncthreads();

        // Manually unrolled inner loop (unroll factor of 4, since BLOCK_SIZE==32)
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; k += 4) {
            value += s_A[ty][k]     * s_B[k][tx];
            value += s_A[ty][k + 1] * s_B[k + 1][tx];
            value += s_A[ty][k + 2] * s_B[k + 2][tx];
            value += s_A[ty][k + 3] * s_B[k + 3][tx];
        }

        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = value;
    }
}

// C++ interface
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be of same size");

    int N = A.size(0);
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, A.get_device());
    auto C = torch::zeros({N, N}, options);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matmul_unroll_kernel<<<blocks, threads>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), N);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix Multiplication (CUDA) with manual loop unrolling");
}
