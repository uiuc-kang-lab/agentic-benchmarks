#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")

// Warp-level reduction using shuffle intrinsics
__inline__ __device__ float warpReduceSum(float val) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xffffffff, val, offset);
    return val;
}

// Device function to compute partial sum and partial sum of squares
__device__ void computePartialSums(const float* __restrict__ input,
                                     int c, int N, int C, int H, int W,
                                     int tid, int stride,
                                     float &partialSum, float &partialSumSq) {
    int numElements = N * H * W;
    partialSum = 0.0f;
    partialSumSq = 0.0f;
    for (int i = tid; i < numElements; i += stride) {
        int n = i / (H * W);
        int r = i % (H * W);
        int h = r / W;
        int w = r % W;
        int idx = ((n * C + c) * H + h) * W + w;
        float val = input[idx];
        partialSum += val;
        partialSumSq += val * val;
    }
}

// Block-level reduction using warp-level primitives and shared memory
__device__ void blockReduceSum(float &sum, float &sumSq) {
    int lane = threadIdx.x % warpSize;
    int warpId = threadIdx.x / warpSize;

    float sum_val = warpReduceSum(sum);
    float sumSq_val = warpReduceSum(sumSq);

    __shared__ float sharedSum[32]; // assuming maximum 32 warps per block
    __shared__ float sharedSumSq[32];

    if(lane == 0) {
        sharedSum[warpId] = sum_val;
        sharedSumSq[warpId] = sumSq_val;
    }
    __syncthreads();

    // Final reduction performed by thread 0
    if(threadIdx.x == 0) {
        float totalSum = 0.0f;
        float totalSumSq = 0.0f;
        int numWarps = (blockDim.x + warpSize - 1) / warpSize;
        for (int i = 0; i < numWarps; i++) {
            totalSum += sharedSum[i];
            totalSumSq += sharedSumSq[i];
        }
        sum = totalSum;
        sumSq = totalSumSq;
    }
    __syncthreads();
}

// Device function to normalize a value
__device__ inline float normalizeValue(float val, float mean, float invStd, float w, float b) {
    return (val - mean) * invStd * w + b;
}

// Kernel using modular device functions with adaptive block size
__global__ void adaptive_blocksize_batch_norm_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ running_mean,
    float* __restrict__ running_var,
    bool training,
    float momentum,
    float eps,
    float* __restrict__ output,
    int N,
    int C,
    int H,
    int W) {

    int c = blockIdx.x;  // each block handles one channel
    int tid = threadIdx.x;
    int stride = blockDim.x;
    int numElements = N * H * W;

    // Phase 1: Compute partial sums for the channel
    float partialSum, partialSumSq;
    computePartialSums(input, c, N, C, H, W, tid, stride, partialSum, partialSumSq);
    blockReduceSum(partialSum, partialSumSq);

    __shared__ float stats[2]; // shared memory for channel mean and variance
    float mean, var;
    if (tid == 0) {
        mean = partialSum / numElements;
        var = partialSumSq / numElements - mean * mean;
        if (training) {
            running_mean[c] = (1 - momentum) * running_mean[c] + momentum * mean;
            running_var[c] = (1 - momentum) * running_var[c] + momentum * var;
        } else {
            mean = running_mean[c];
            var = running_var[c];
        }
        stats[0] = mean;
        stats[1] = var;
    }
    __syncthreads();
    mean = stats[0];
    var = stats[1];

    float invStd = rsqrtf(var + eps);
    float channelWeight = weight[c];
    float channelBias = bias[c];

    // Phase 2: Normalize and write output
    for (int i = tid; i < numElements; i += stride) {
        int n = i / (H * W);
        int r = i % (H * W);
        int h = r / W;
        int w = r % W;
        int idx = ((n * C + c) * H + h) * W + w;
        float val = input[idx];
        output[idx] = normalizeValue(val, mean, invStd, channelWeight, channelBias);
    }
}

// Host function called from PyTorch
torch::Tensor adaptive_forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    torch::Tensor running_mean,
    torch::Tensor running_var,
    bool training,
    float momentum,
    float eps) {

    CHECK_CUDA(input);
    CHECK_CUDA(weight);
    CHECK_CUDA(bias);
    CHECK_CUDA(running_mean);
    CHECK_CUDA(running_var);

    CHECK_CONTIGUOUS(input);
    CHECK_CONTIGUOUS(weight);
    CHECK_CONTIGUOUS(bias);
    CHECK_CONTIGUOUS(running_mean);
    CHECK_CONTIGUOUS(running_var);

    int N = input.size(0);
    int C = input.size(1);
    int H = input.size(2);
    int W = input.size(3);

    auto output = torch::empty_like(input);

    int threads = 512; // Experiment with different block sizes to find optimal
    size_t shared_mem = 0; // Static shared memory usage via __shared__ declarations

    adaptive_blocksize_batch_norm_kernel<<<C, threads, shared_mem>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.data_ptr<float>(),
        running_mean.data_ptr<float>(),
        running_var.data_ptr<float>(),
        training,
        momentum,
        eps,
        output.data_ptr<float>(),
        N, C, H, W
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &adaptive_forward_cuda, "Adaptive Block Size BatchNorm forward (CUDA)");
}
