#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void relu_kernel_2d(
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ input,
    const int64_t size) {
    
    // 2D grid for better occupancy
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    const int block_size = blockDim.x * blockDim.y;
    const int bid = blockIdx.x + blockIdx.y * gridDim.x;
    const int idx = tid + bid * block_size;
    
    if (idx < size) {
        output[idx] = input[idx] > 0 ? input[idx] : 0;
    }
}

torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    
    // Use 16x16 thread blocks
    dim3 threads(16, 16);
    const int total_threads = threads.x * threads.y;
    
    // Calculate grid dimensions to precisely cover the elements
    const int64_t num_elements = input.numel();
    const int blocks_x = (num_elements + total_threads - 1) / total_threads;
    // Use a reasonable number of blocks in y-dimension for better load distribution
    const int blocks_y = (blocks_x + 31) / 32;  // Limit y-dimension, adjust x accordingly
    const int final_blocks_x = (blocks_x + blocks_y - 1) / blocks_y;
    dim3 blocks(final_blocks_x, blocks_y);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "relu_kernel_2d", ([&] {
        relu_kernel_2d<scalar_t><<<blocks, threads>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            input.numel()
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ReLU forward (CUDA)");
}