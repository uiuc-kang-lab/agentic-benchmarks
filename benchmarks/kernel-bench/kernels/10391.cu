#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

__device__ __forceinline__ float compute_inner(float x) {
    const float sqrt_2_over_pi = 0.7978845608f;
    const float coeff = 0.044715f;
    float x_cubed = x * x * x;
    return (x + coeff * x_cubed) * sqrt_2_over_pi;
}

__device__ __forceinline__ float compute_gelu(float x) {
    float inner = compute_inner(x);
    float tanh_val = __tanhf(inner);
    return 0.5f * x * (1.0f + tanh_val);
}

__global__ void gelu_kernel(const float* x, float* y, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = compute_gelu(x[i]);
    }
}

torch::Tensor gelu_forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(x.is_contiguous(), "Input tensor must be contiguous");
    
    auto y = torch::empty_like(x);
    int n = x.numel();
    
    const int threads = 256;
    int blocks = (n + threads - 1) / threads;
    
    gelu_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        n
    );
    
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &gelu_forward, "GELU forward CUDA implementation");
}