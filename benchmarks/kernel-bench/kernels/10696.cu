#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// This kernel uses __ldg() to optimize global memory reads by taking advantage of the read-only cache
// and assumes that the input pointer is 128-bit aligned. It performs a reverse cumulative sum along the last dimension
// using warp-level shuffles and computes per-warp offsets in shared memory. Global stores are coalesced and
// the kernel is designed to work when processing contiguous rows (n <= 1024).

template <typename scalar_t>
__global__ void aligned_ldg_reverse_cumsum_kernel(const scalar_t* __restrict__ input,
                                                    scalar_t* __restrict__ output,
                                                    int64_t n) {
    // Each block handles one row (outer dimension)
    int row = blockIdx.x;
    const int64_t row_offset = row * n;

    int tid = threadIdx.x;
    int lane = tid & 31;  // Lane index within the warp

    // Load element in reverse order using __ldg() for read-only global memory load
    scalar_t val = 0;
    if (tid < n) {
        // __ldg() instructs the compiler to use read-only cache and assumes the pointer is 128-bit aligned
        val = __ldg(&input[row_offset + (n - 1 - tid)]);
    }

    // Perform warp-level inclusive scan using shuffle intrinsics
    for (int offset = 1; offset < 32; offset *= 2) {
        scalar_t tmp = __shfl_up_sync(0xffffffff, val, offset);
        if (lane >= offset) {
            val += tmp;
        }
    }

    // Each warp's last active thread writes its result to shared memory
    __shared__ scalar_t warp_sums[32];
    int warp_id = tid >> 5;
    if (tid < n && (lane == 31 || tid == n - 1)) {
        warp_sums[warp_id] = val;
    }
    __syncthreads();

    // Each thread computes the offset for its warp by summing the totals of previous warps
    if (tid < n) {
        scalar_t warp_offset = 0;
        for (int w = 0; w < warp_id; w++) {
            warp_offset += warp_sums[w];
        }
        scalar_t final_val = val + warp_offset;
        // Write the computed cumulative sum back in the original order
        output[row_offset + (n - 1 - tid)] = final_val;
    }
}

at::Tensor reverse_cumsum(at::Tensor x, int64_t dim) {
    // Ensure the tensor is contiguous and on CUDA
    x = x.contiguous();
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    
    const int ndim = x.dim();
    TORCH_CHECK(dim >= 0 && dim < ndim, "dim out of range");

    auto output = at::empty_like(x);

    int64_t n = x.size(dim);
    int64_t outer = x.numel() / n;

    // Use the optimized kernel only if the cum-sum is performed along the last dimension and n is not large
    if (dim == ndim - 1 && n <= 1024) {
        // Determine number of threads as next power of 2 >= n (capped at 1024)
        int threads = 1;
        while (threads < n) {
            threads *= 2;
        }
        if (threads > 1024) threads = 1024;

        dim3 blocks(outer);
        dim3 threadsPerBlock(threads);

        AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "aligned_ldg_reverse_cumsum_kernel", ([&] {
            aligned_ldg_reverse_cumsum_kernel<scalar_t><<<blocks, threadsPerBlock>>>(
                x.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                n);
        }));
    } else {
        // Fallback to the flip-cumsum-flip method for non-ideal cases
        auto x_flipped = x.flip(dim);
        auto cumsum = x_flipped.cumsum(dim);
        output = cumsum.flip(dim);
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &reverse_cumsum, "Reverse cumulative sum optimized with __ldg() for aligned accesses (CUDA)");
}
