#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device function to compute KL divergence for a single element
__device__ __forceinline__ float compute_kl_div(float log_pred, float target) {
    return expf(log_pred) - target * log_pred;
}

// Device function for warp-level reduction
__device__ __forceinline__ float warp_reduce_sum(float val) {
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

// Device function for block-level reduction
__device__ __forceinline__ void block_reduce_sum(float* shared_data, int tid) {
    for (int stride = blockDim.x/2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }
    
    // Final warp reduction
    if (tid < 32) {
        float warp_sum = shared_data[tid];
        if (tid + 32 < blockDim.x) {
            warp_sum += shared_data[tid + 32];
        }
        warp_sum = warp_reduce_sum(warp_sum);
        shared_data[tid] = warp_sum;
    }
}

__global__ void kl_div_kernel_modular_optimized(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    // Get thread indices
    const int tid = threadIdx.x;
    const int gid = blockIdx.x * blockDim.x + tid;
    
    // Shared memory for partial sums
    extern __shared__ float partial_sums[];
    
    float sum = 0.0f;
    
    // Calculate KL divergence for this thread's elements
    #pragma unroll 4
    for (int i = gid; i < n; i += blockDim.x * gridDim.x) {
        sum += compute_kl_div(log_predictions[i], targets[i]);
    }
    
    // Store in shared memory
    partial_sums[tid] = sum;
    __syncthreads();
    
    // Perform block-level reduction
    block_reduce_sum(partial_sums, tid);
    
    // Write result for this block to global memory
    if (tid == 0) {
        atomicAdd(output, partial_sums[0]);
    }
}

torch::Tensor kl_div_cuda_forward_modular_optimized(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    const int threads = 256;
    const int blocks = min((n + threads - 1) / threads, 1024);
    const int shared_mem = threads * sizeof(float);
    
    kl_div_kernel_modular_optimized<<<blocks, threads, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward_modular_optimized, "KL divergence forward with modular optimizations (CUDA)");
}