#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void sigmoid_kernel(const scalar_t* __restrict__ input,
                               scalar_t* __restrict__ output,
                               const int64_t size) {
  const int stride = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += stride) {
    float val = static_cast<float>(-input[i]);
    float exp_val = expf(val);
    float r = 1.0f / (1.0f + exp_val);
    output[i] = static_cast<scalar_t>(r);
  }
}

torch::Tensor forward(torch::Tensor input) {
  auto output = torch::empty_like(input);
  const int64_t size = input.numel();

  const int threads = 256;
  const int max_blocks = 65535;  // Maximum blocks per grid dimension
  const int min_blocks = (size + threads - 1) / threads;
  const int blocks = min(max_blocks, min_blocks);

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sigmoid_kernel", [&] {
    const auto* input_data = input.data_ptr<scalar_t>();
    auto* output_data = output.data_ptr<scalar_t>();

    sigmoid_kernel<scalar_t><<<blocks, threads>>>(input_data, output_data, size);
  });

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "Sigmoid forward (CUDA)");
}