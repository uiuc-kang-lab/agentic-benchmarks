#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

template <typename scalar_t>
__global__ void masked_cumsum_kernel(
    const scalar_t* __restrict__ x,
    const bool* __restrict__ mask,
    scalar_t* __restrict__ output,
    int64_t N,
    int64_t L) {

    extern __shared__ char shared_mem[];
    scalar_t* s_data = (scalar_t*)shared_mem;
    bool* s_mask = (bool*)(s_data + L);

    const int row = blockIdx.x;
    const int tid = threadIdx.x;
    
    if (row >= N) return;

    const scalar_t* x_row = x + row * L;
    const bool* mask_row = mask + row * L;
    scalar_t* output_row = output + row * L;

    // Cooperatively load data into shared memory
    for (int i = tid; i < L; i += BLOCK_SIZE) {
        s_data[i] = x_row[i];
        s_mask[i] = mask_row[i];
    }
    __syncthreads();

    // Compute cumulative sum
    scalar_t sum = 0;
    for (int i = 0; i < L; i++) {
        if (s_mask[i]) {
            sum += s_data[i];
        }
        if (tid == 0) {
            output_row[i] = sum;
        }
    }
}

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor masked_cumsum(
    const torch::Tensor& x,
    const torch::Tensor& mask,
    int64_t dim) {

    CHECK_INPUT(x);
    CHECK_INPUT(mask);
    TORCH_CHECK(x.sizes() == mask.sizes(), "x and mask must have the same shape");
    TORCH_CHECK(mask.scalar_type() == torch::kBool, "mask must be a boolean tensor");

    if (dim < 0) {
        dim += x.dim();
    }
    TORCH_CHECK(dim >= 0 && dim < x.dim(), "Invalid dimension");

    std::vector<int64_t> perm;
    for (int64_t i = 0; i < x.dim(); ++i) {
        if (i != dim)
            perm.push_back(i);
    }
    perm.push_back(dim);

    auto x_permuted = x.permute(perm).contiguous();
    auto mask_permuted = mask.permute(perm).contiguous();

    int64_t N = x_permuted.numel() / x_permuted.size(-1);
    int64_t L = x_permuted.size(-1);

    auto x_flat = x_permuted.view({N, L});
    auto mask_flat = mask_permuted.view({N, L});
    auto output_flat = torch::empty_like(x_flat);

    const int threads = BLOCK_SIZE;
    const int blocks = N;
    const size_t shared_mem_size = L * (sizeof(typename std::conditional<std::is_same<scalar_t, float>::value, float, double>::type) + sizeof(bool));

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "masked_cumsum_cuda", ([&] {
        masked_cumsum_kernel<scalar_t><<<blocks, threads, shared_mem_size>>>(
            x_flat.data_ptr<scalar_t>(),
            mask_flat.data_ptr<bool>(),
            output_flat.data_ptr<scalar_t>(),
            N,
            L
        );
    }));

    auto output_permuted = output_flat.view(x_permuted.sizes());
    std::vector<int64_t> inv_perm(perm.size());
    for (size_t i = 0; i < perm.size(); ++i) {
        inv_perm[perm[i]] = i;
    }
    auto output = output_permuted.permute(inv_perm);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &masked_cumsum, "Masked Cumulative Sum (CUDA)");
}