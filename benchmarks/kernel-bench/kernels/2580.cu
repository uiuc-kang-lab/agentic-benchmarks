#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized CUDA kernel for ReLU activation
template <typename scalar_t>
__global__ void relu_kernel_optimized(
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ input,
    const int64_t size) {
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < size; i += stride) {
        output[i] = input[i] > 0 ? input[i] : 0;
    }
}

// PyTorch wrapper function
torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    
    const int threads = 256;
    const int blocks = (input.numel() + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "relu_kernel_optimized", ([&] {
        relu_kernel_optimized<scalar_t><<<blocks, threads>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            input.numel()
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ReLU forward optimized (CUDA)");
}