#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 256
#define WARP_SIZE 32

__device__ __forceinline__ float compute_kl_div(float log_pred, float target) {
    return __expf(log_pred) - target * log_pred;
}

__device__ __forceinline__ float warp_reduce_sum(float val) {
    #pragma unroll
    for (int offset = WARP_SIZE/2; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__global__ void efficient_kl_div_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    extern __shared__ float shared_mem[];
    float* partial_sums = shared_mem;
    
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int num_threads = blockDim.x;
    const int warp_id = tid / WARP_SIZE;
    const int lane_id = tid % WARP_SIZE;
    
    float thread_sum = 0.0f;
    
    // Process elements with stride of complete warps
    for (int i = bid * num_threads + tid; i < n; i += gridDim.x * num_threads) {
        if (i < n) {
            float log_pred = log_predictions[i];
            float target = targets[i];
            thread_sum += compute_kl_div(log_pred, target);
        }
    }
    
    // Warp-level reduction
    thread_sum = warp_reduce_sum(thread_sum);
    
    // Store warp results
    if (lane_id == 0) {
        partial_sums[warp_id] = thread_sum;
    }
    __syncthreads();
    
    // Final reduction by first warp
    if (warp_id == 0 && lane_id < (num_threads / WARP_SIZE)) {
        float sum = partial_sums[lane_id];
        sum = warp_reduce_sum(sum);
        
        if (lane_id == 0) {
            atomicAdd(output, sum);
        }
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    // Launch parameters
    const int threads = TILE_SIZE;
    const int blocks = min((n + threads - 1) / threads, 1024);
    const int shared_mem = (threads / WARP_SIZE) * sizeof(float);
    
    efficient_kl_div_kernel<<<blocks, threads, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward (CUDA)");
}