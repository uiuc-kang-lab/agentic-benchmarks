#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

__constant__ int constant_kernel_size;
__constant__ int constant_stride;
__constant__ int constant_padding;
__constant__ int constant_dilation;

// Max Pool 2D using constant memory
template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* input,
    scalar_t* output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width
) {
    const int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    const int ow = output_idx % output_width;
    const int oh = (output_idx / output_width) % output_height;
    const int c = (output_idx / (output_width * output_height)) % channels;
    const int b = output_idx / (output_width * output_height * channels);

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    #pragma unroll
    for (int kh = 0; kh < constant_kernel_size; kh++) {
        #pragma unroll
        for (int kw = 0; kw < constant_kernel_size; kw++) {
            const int ih = oh * constant_stride - constant_padding + kh * constant_dilation;
            const int iw = ow * constant_stride - constant_padding + kw * constant_dilation;

            if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
                const int input_idx = b * (channels * input_height * input_width) +
                                    c * (input_height * input_width) +
                                    ih * input_width +
                                    iw;
                max_val = max(max_val, input[input_idx]);
            }
        }
    }

    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    int h_kernel_size = kernel_size;
    int h_stride = stride;
    int h_padding = padding;
    int h_dilation = dilation;

    hipMemcpyToSymbol(HIP_SYMBOL(constant_kernel_size), &h_kernel_size, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(constant_stride), &h_stride, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(constant_padding), &h_padding, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(constant_dilation), &h_dilation, sizeof(int));

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}