#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// CUDA kernel for computing sum of squares using shared memory and warp-level primitives
__global__ void compute_norm_kernel_atomic_optimized(const float* input, float* norm_out, int numel) {
    __shared__ float shared_sum[256];
    
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize shared memory
    float sum = 0.0f;
    
    // Compute partial sums
    while (idx < numel) {
        sum += input[idx] * input[idx];
        idx += blockDim.x * gridDim.x;
    }
    shared_sum[tid] = sum;
    __syncthreads();
    
    // Reduce within block
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            shared_sum[tid] += shared_sum[tid + stride];
        }
        __syncthreads();
    }
    
    // Warp-level reduction
    if (tid < 32) {
        float val = shared_sum[tid];
        for (int offset = 16; offset > 0; offset >>= 1) {
            val += __shfl_down_sync(0xffffffff, val, offset);
        }
        if (tid == 0) {
            atomicAdd(norm_out, val);
        }
    }
}

// CUDA kernel for normalization
__global__ void normalize_kernel(const float* input, float* output, 
                               float norm, int numel) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < numel) {
        output[idx] = input[idx] / norm;
    }
}

torch::Tensor forward(torch::Tensor input) {
    // Input validation
    TORCH_CHECK(input.is_cuda(), "Input tensor must be on GPU");
    TORCH_CHECK(input.is_contiguous(), "Input tensor must be contiguous");
    TORCH_CHECK(input.scalar_type() == torch::kFloat32, "Input must be float32");

    // Create output tensor with same shape as input
    auto output = torch::empty_like(input);
    auto norm_tensor = torch::zeros({1}, input.options());
    
    // Get raw pointers
    const float* input_ptr = input.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();
    float* norm_ptr = norm_tensor.data_ptr<float>();
    
    // Calculate total number of elements
    int numel = input.numel();
    
    // Calculate grid and block dimensions
    const int threads = 256;
    const int blocks = min(65535, (numel + threads - 1) / threads);

    // First kernel: compute sum of squares
    compute_norm_kernel_atomic_optimized<<<blocks, threads>>>(input_ptr, norm_ptr, numel);
    
    // Get norm value and compute square root
    float norm_val;
    hipMemcpy(&norm_val, norm_ptr, sizeof(float), hipMemcpyDeviceToHost);
    norm_val = sqrt(norm_val);
    
    // Second kernel: normalize the tensor
    normalize_kernel<<<blocks, threads>>>(input_ptr, output_ptr, norm_val, numel);
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Frobenius norm normalization");
}