#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void matmul_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    int bx = blockIdx.x * TILE_SIZE;
    int by = blockIdx.y * TILE_SIZE;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by + ty;
    int col = bx + tx;
    
    float sum = 0.0f;
    
    for (int i = 0; i < K; i += TILE_SIZE) {
        if (row < M && (i + tx) < K)
            As[ty][tx] = A[row * K + i + tx];
        else
            As[ty][tx] = 0.0f;
            
        if (col < N && (i + ty) < K)
            Bs[ty][tx] = B[(i + ty) * N + col];
        else
            Bs[ty][tx] = 0.0f;
            
        __syncthreads();
        
        #pragma unroll 4
        for (int k = 0; k < TILE_SIZE; k++)
            sum = __fmaf_rn(As[ty][k], Bs[k][tx], sum);
            
        __syncthreads();
    }
    
    if (row < M && col < N)
        C[row * N + col] = sum;
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.is_contiguous(), "A must be contiguous");
    TORCH_CHECK(B.is_contiguous(), "B must be contiguous");
    
    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);
    
    torch::Tensor C = torch::zeros({M, N}, A.options());
    
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE,
                   (M + TILE_SIZE - 1) / TILE_SIZE);
    
    matmul_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        M, N, K
    );
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication (CUDA)");
}