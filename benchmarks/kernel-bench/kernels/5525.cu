#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int WARP_SIZE = 32;
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    const int total_width = output_width * channels;
    const int warp_id = tid / WARP_SIZE;
    const int lane_id = tid % WARP_SIZE;
    
    const int elements_per_warp = WARP_SIZE;
    const int total_warps = (batch_size * output_height * total_width + elements_per_warp - 1) / elements_per_warp;
    
    if (warp_id >= total_warps) return;

    const int warp_offset = warp_id * elements_per_warp;
    const int global_width_idx = warp_offset % total_width + lane_id;
    const int global_height_idx = (warp_offset / total_width) % output_height;
    const int batch_idx = warp_offset / (total_width * output_height);

    if (batch_idx >= batch_size || global_width_idx >= total_width) return;

    const int channel_idx = global_width_idx / output_width;
    const int width_idx = global_width_idx % output_width;

    if (channel_idx >= channels) return;

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    const int input_batch_offset = batch_idx * channels * input_height * input_width;
    const int input_channel_offset = channel_idx * input_height * input_width;

    #pragma unroll
    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = global_height_idx * stride - padding + kh * dilation;
        if (ih >= 0 && ih < input_height) {
            const int input_h_offset = ih * input_width;
            
            #pragma unroll
            for (int kw = 0; kw < kernel_size; kw++) {
                const int iw = width_idx * stride - padding + kw * dilation;
                if (iw >= 0 && iw < input_width) {
                    const int input_idx = input_batch_offset + 
                                        input_channel_offset + 
                                        input_h_offset + 
                                        iw;
                    max_val = max(max_val, input[input_idx]);
                }
            }
        }
    }

    const int output_idx = batch_idx * (channels * output_height * output_width) +
                          channel_idx * (output_height * output_width) +
                          global_height_idx * output_width +
                          width_idx;
    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int threads = 256;
    const int total_elements = batch_size * channels * output_height * output_width;
    const int blocks = (total_elements + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}