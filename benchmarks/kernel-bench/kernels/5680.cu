#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <limits>

// Combined max pooling kernel: Uses grid-stride loops for robustness and compile-time unrolling for common kernel sizes.
// When static_kernel_size > 0 (e.g., 2 or 3), the loops are unrolled and __ldg() is used for improved memory access.
// Otherwise, the dynamic path (static_kernel_size == -1) is used to support arbitrary kernel sizes.

template <typename scalar_t, int static_kernel_size>
__global__ void max_pool2d_combined_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,  // used only when static_kernel_size == -1
    const int stride,
    const int padding,
    const int dilation
) {
    const int total = batch_size * channels * output_height * output_width;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int gridSize = blockDim.x * gridDim.x;

    while (index < total) {
        // Compute 4D indices from the flat index
        int ow = index % output_width;
        int oh = (index / output_width) % output_height;
        int c  = (index / (output_width * output_height)) % channels;
        int b  = index / (output_width * output_height * channels);

        // Initialize maximum value to negative infinity
        scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
        // Base offset for the input feature map for this batch and channel
        int input_offset = (b * channels + c) * input_height * input_width;

        if (static_kernel_size > 0) {
            // Compile-time known kernel size allows loop unrolling for better performance
            #pragma unroll
            for (int kh = 0; kh < static_kernel_size; kh++) {
                int ih = oh * stride - padding + kh * dilation;
                if (ih >= 0 && ih < input_height) {
                    #pragma unroll
                    for (int kw = 0; kw < static_kernel_size; kw++) {
                        int iw = ow * stride - padding + kw * dilation;
                        if (iw >= 0 && iw < input_width) {
                            scalar_t val = __ldg(&input[input_offset + ih * input_width + iw]);
                            if (val > max_val)
                                max_val = val;
                        }
                    }
                }
            }
        } else {
            // Dynamic kernel size: iterate based on the runtime provided kernel_size
            for (int kh = 0; kh < kernel_size; kh++) {
                int ih = oh * stride - padding + kh * dilation;
                if (ih >= 0 && ih < input_height) {
                    for (int kw = 0; kw < kernel_size; kw++) {
                        int iw = ow * stride - padding + kw * dilation;
                        if (iw >= 0 && iw < input_width) {
                            scalar_t val = input[input_offset + ih * input_width + iw];
                            if (val > max_val)
                                max_val = val;
                        }
                    }
                }
            }
        }
        
        output[index] = max_val;
        index += gridSize;
    }
}

// Host function for launching the combined CUDA kernel
// This function calculates the output dimensions and dispatches the kernel with either compile-time kernel size or dynamic kernel size.

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const int batch_size = input.size(0);
    const int channels = input.size(1);
    const int input_height = input.size(2);
    const int input_width = input.size(3);

    const int output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const int output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int total_elements = batch_size * channels * output_height * output_width;
    const int threads = 256;
    const int blocks = (total_elements + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool2d_cuda_forward", ([&] {
        // Use compile-time unrolling for common small kernel sizes (2 and 3), otherwise fall back to the dynamic kernel
        if (kernel_size == 2) {
            max_pool2d_combined_kernel<scalar_t, 2><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size,
                channels,
                input_height,
                input_width,
                output_height,
                output_width,
                kernel_size, // not used in this branch
                stride,
                padding,
                dilation
            );
        } else if (kernel_size == 3) {
            max_pool2d_combined_kernel<scalar_t, 3><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size,
                channels,
                input_height,
                input_width,
                output_height,
                output_width,
                kernel_size,
                stride,
                padding,
                dilation
            );
        } else {
            max_pool2d_combined_kernel<scalar_t, -1><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size,
                channels,
                input_height,
                input_width,
                output_height,
                output_width,
                kernel_size,
                stride,
                padding,
                dilation
            );
        }
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Combined efficient Max Pool 2D forward (CUDA)");
}
