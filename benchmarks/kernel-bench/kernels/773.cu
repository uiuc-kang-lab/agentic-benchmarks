#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void optimized_matmul_kernel(const float* __restrict__ A, 
                                        const float* __restrict__ B, 
                                        float* __restrict__ C, 
                                        const int M, const int N, const int K) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;
    float sum = 0.0f;

    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        int tiledCol = t * TILE_SIZE + tx;
        int tiledRow = t * TILE_SIZE + ty;

        tileA[ty][tx] = (row < M && tiledCol < K) ? A[row * K + tiledCol] : 0.0f;
        tileB[ty][tx] = (tiledRow < K && col < N) ? B[tiledRow * N + col] : 0.0f;
        
        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += tileA[ty][k] * tileB[k][tx];
        }
        __syncthreads();
    }

    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.is_contiguous(), "A must be contiguous");
    TORCH_CHECK(B.is_contiguous(), "B must be contiguous");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);

    optimized_matmul_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        M, N, K
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Matrix Multiplication (CUDA)");
}