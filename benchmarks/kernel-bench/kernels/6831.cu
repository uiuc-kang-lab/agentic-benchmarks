#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <cfloat>

__global__ void atomic_optimized_argmax_kernel(
    const float* __restrict__ x,
    int64_t* __restrict__ indices,
    const int dimSize,
    const int innerSize) {

    int global_idx = blockIdx.x;
    int outer_idx = global_idx / innerSize;
    int inner_idx = global_idx % innerSize;
    int base_offset = outer_idx * dimSize * innerSize + inner_idx;

    extern __shared__ float shared[];
    int* sidx = (int*)&shared[blockDim.x];

    float thread_max = -FLT_MAX;
    int thread_max_idx = 0;

    for (int i = threadIdx.x; i < dimSize; i += blockDim.x) {
        float val = __ldg(&x[base_offset + i * innerSize]);
        if (val > thread_max) {
            thread_max = val;
            thread_max_idx = i;
        }
    }

    shared[threadIdx.x] = thread_max;
    sidx[threadIdx.x] = thread_max_idx;
    __syncthreads();

    // Use warp-level primitives for intra-warp reduction
    for (int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (threadIdx.x < s) {
            if (shared[threadIdx.x + s] > shared[threadIdx.x]) {
                shared[threadIdx.x] = shared[threadIdx.x + s];
                sidx[threadIdx.x] = sidx[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    // Warp-level reduction (assumes blockDim.x == 128 for simplicity in this example)
    if (threadIdx.x < 32) { 
        volatile float* vshared = shared;
        volatile int* vsidx = sidx;
        for (int offset = 32; offset > 0; offset /= 2) {
            if (vshared[threadIdx.x + offset] > vshared[threadIdx.x]) {
                vshared[threadIdx.x] = vshared[threadIdx.x + offset];
                vsidx[threadIdx.x] = vsidx[threadIdx.x + offset];
            }
        }
    }

    if (threadIdx.x == 0) {
        atomicMax(&indices[global_idx], sidx[0]);
    }
}

torch::Tensor atomic_optimized_argmax_forward_cuda(const torch::Tensor& x, const int64_t dim) {
    TORCH_CHECK(x.scalar_type() == at::kFloat, "Only float32 supported.");
    auto x_contig = x.contiguous();
    auto sizes = x_contig.sizes();
    int ndim = x_contig.dim();
    TORCH_CHECK(dim >= 0 && dim < ndim, "Invalid dim.");

    int outerSize = 1;
    for (int d = 0; d < dim; d++) outerSize *= sizes[d];
    int dimSize = sizes[dim];
    int innerSize = 1;
    for (int d = dim + 1; d < ndim; d++) innerSize *= sizes[d];

    std::vector<int64_t> out_sizes;
    for (int d = 0; d < ndim; d++) if (d != dim) out_sizes.push_back(sizes[d]);
    auto indices = torch::empty(out_sizes, torch::TensorOptions().device(x.device()).dtype(torch::kLong));

    int blockSize = 128; // Kept as 128 based on performance tuning hints
    dim3 grid(outerSize * innerSize);
    size_t shared_mem = blockSize * (sizeof(float) + sizeof(int));

    atomic_optimized_argmax_kernel<<<grid, blockSize, shared_mem>>>(
        x_contig.data_ptr<float>(),
        indices.data_ptr<int64_t>(),
        dimSize,
        innerSize
    );

    return indices;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &atomic_optimized_argmax_forward_cuda, "Atomic Optimized ArgMax CUDA forward");
}