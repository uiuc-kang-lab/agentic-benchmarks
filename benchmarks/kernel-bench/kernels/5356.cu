#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Define constant memory for kernel parameters
__constant__ int const_params[4]; // [kernel_size, stride, padding, dilation]

template <typename scalar_t>
__global__ void max_pool2d_constant_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width
) {
    const int kernel_size = const_params[0];
    const int stride = const_params[1];
    const int padding = const_params[2];
    const int dilation = const_params[3];

    const int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    const int ow = output_idx % output_width;
    const int oh = (output_idx / output_width) % output_height;
    const int c = (output_idx / (output_width * output_height)) % channels;
    const int b = output_idx / (output_width * output_height * channels);

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    for (int kh = 0; kh < kernel_size; kh++) {
        for (int kw = 0; kw < kernel_size; kw++) {
            const int ih = oh * stride - padding + kh * dilation;
            const int iw = ow * stride - padding + kw * dilation;

            if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
                const int input_idx = b * channels * input_height * input_width +
                                      c * input_height * input_width +
                                      ih * input_width +
                                      iw;
                
                // Use __ldg to load data for read-only access
                scalar_t val = __ldg(&input[input_idx]);
                max_val = max(max_val, val);
            }
        }
    }

    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_constant_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    int params[4] = {kernel_size, stride, padding, dilation};
    hipMemcpyToSymbol(HIP_SYMBOL(const_params), params, sizeof(params));

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "max_pool2d_constant_cuda_forward", ([&] {
        max_pool2d_constant_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_constant_cuda_forward, "Optimized Max Pool 2D forward with constant memory (CUDA)");
}