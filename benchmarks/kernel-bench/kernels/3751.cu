#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Define constants in constant memory for fast broadcast access
__constant__ float UPPER_THRESHOLD = 20.0f;
__constant__ float LOWER_THRESHOLD = -20.0f;

template <typename scalar_t>
__global__ void softplus_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < size) {
        const scalar_t x = input[idx];
        if (x > UPPER_THRESHOLD) {
            output[idx] = x;
        } else if (x < LOWER_THRESHOLD) {
            output[idx] = exp(x);
        } else {
            output[idx] = log1p(exp(x));
        }
    }
}

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    const int threads = 512;
    const int blocks = (size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "softplus_forward_cuda", ([&] {
        softplus_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}