#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>

// Optimized kernel that combines efficient indexing and asynchronous execution

template <typename scalar_t>
__global__ void optimized_conv_transpose2d_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int out_channels,
    const int kernel_h,
    const int kernel_w,
    const int stride,
    const int padding,
    const int output_padding,
    const int groups,
    const int dilation,
    const int out_height,
    const int out_width
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_elements = batch_size * out_channels * out_height * out_width;
    if (idx >= total_elements) return;

    // Pre-compute channel and spatial indices
    const int ow = idx % out_width;
    const int oh = (idx / out_width) % out_height;
    const int oc = (idx / (out_width * out_height)) % out_channels;
    const int b = idx / (out_width * out_height * out_channels);

    // Pre-compute group-related constants
    const int out_channels_per_group = out_channels / groups;
    const int g = oc / out_channels_per_group;
    const int oc_group = oc % out_channels_per_group;
    const int in_channels_per_group = in_channels / groups;
    const int ic_start = g * in_channels_per_group;

    // Pre-compute input and weight offsets
    const int b_offset = b * in_channels * in_height * in_width;
    const int w_offset = oc_group * (kernel_h * kernel_w);
    
    // Initialize output value
    scalar_t val = (bias != nullptr) ? bias[oc] : static_cast<scalar_t>(0);

    #pragma unroll 4
    for (int kh = 0; kh < kernel_h; ++kh) {
        const int h_in_base = (oh - kh * dilation + padding);
        if (h_in_base % stride != 0) continue;
        const int h_in = h_in_base / stride;
        if (h_in < 0 || h_in >= in_height) continue;

        #pragma unroll 4
        for (int kw = 0; kw < kernel_w; ++kw) {
            const int w_in_base = (ow - kw * dilation + padding);
            if (w_in_base % stride != 0) continue;
            const int w_in = w_in_base / stride;
            if (w_in < 0 || w_in >= in_width) continue;

            const int in_spatial_offset = h_in * in_width + w_in;
            const int w_spatial_offset = kh * kernel_w + kw;

            for (int ic = 0; ic < in_channels_per_group; ++ic) {
                const int in_idx = b_offset + (ic_start + ic) * in_height * in_width + in_spatial_offset;
                const int w_idx = (ic_start + ic) * (out_channels_per_group * kernel_h * kernel_w) + 
                                w_offset + w_spatial_offset;
                
                val += input[in_idx] * weight[w_idx];
            }
        }
    }
    output[idx] = val;
}

// Optimized forward function with asynchronous execution and double buffering

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int output_padding,
    int groups,
    int dilation = 1
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(x.dim() == 4, "Input must be 4D");
    TORCH_CHECK(weight.dim() == 4, "Weight must be 4D");

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);

    const int out_channels = weight.size(1) * groups;
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);

    if (bias.has_value() && bias->defined()) {
        TORCH_CHECK(bias->numel() == out_channels, "Bias must have out_channels elements");
        TORCH_CHECK(bias->device().is_cuda(), "Bias must be a CUDA tensor");
    }

    const int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_h - 1) + output_padding + 1;
    const int out_width = (in_width - 1) * stride - 2 * padding + dilation * (kernel_w - 1) + output_padding + 1;

    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, x.options());

    // Number of elements per image in the input and output
    const int input_batch_elems = in_channels * in_height * in_width;
    const int output_batch_elems = out_channels * out_height * out_width;

    // Determine chunk size for pipelining (tunable parameter)
    int chunk_size = batch_size > 16 ? 16 : batch_size;
    int num_chunks = (batch_size + chunk_size - 1) / chunk_size;

    // Create two CUDA streams for double buffering
    hipStream_t streams[2];
    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "optimized_conv_transpose2d_cuda", ([&] {
        using scalar_t_ = scalar_t;
        size_t in_chunk_bytes = input_batch_elems * sizeof(scalar_t_);
        size_t out_chunk_bytes = output_batch_elems * sizeof(scalar_t_);

        // Allocate double buffers for input and output chunks on device
        scalar_t_* d_temp_input[2];
        scalar_t_* d_temp_output[2];
        hipMalloc(&d_temp_input[0], chunk_size * in_chunk_bytes);
        hipMalloc(&d_temp_input[1], chunk_size * in_chunk_bytes);
        hipMalloc(&d_temp_output[0], chunk_size * out_chunk_bytes);
        hipMalloc(&d_temp_output[1], chunk_size * out_chunk_bytes);

        for (int i = 0; i < num_chunks; i++) {
            int current_chunk = std::min(chunk_size, batch_size - i * chunk_size);
            int stream_idx = i % 2;
            hipStream_t stream = streams[stream_idx];

            // Asynchronously copy the input chunk from the global input tensor to the temporary buffer
            const scalar_t_* src_ptr = reinterpret_cast<const scalar_t_*>(x.data_ptr<scalar_t_>()) + i * chunk_size * input_batch_elems;
            hipMemcpyAsync(d_temp_input[stream_idx], src_ptr,
                            current_chunk * in_chunk_bytes,
                            hipMemcpyDeviceToDevice, stream);

            // Launch the convolution kernel on the temporary input buffer
            const int total_output_elems = current_chunk * out_channels * out_height * out_width;
            int threads_ = 256;
            int blocks_ = (total_output_elems + threads_ - 1) / threads_;
            optimized_conv_transpose2d_kernel<scalar_t_><<<blocks_, threads_, 0, stream>>>(
                d_temp_input[stream_idx],
                weight.data_ptr<scalar_t_>(),
                (bias.has_value() && bias->defined()) ? bias->data_ptr<scalar_t_>() : nullptr,
                d_temp_output[stream_idx],
                current_chunk,
                in_channels,
                in_height,
                in_width,
                out_channels,
                kernel_h,
                kernel_w,
                stride,
                padding,
                output_padding,
                groups,
                dilation,
                out_height,
                out_width
            );

            // Asynchronously copy the resulting output chunk to the final output tensor
            scalar_t_* dst_ptr = reinterpret_cast<scalar_t_*>(output.data_ptr<scalar_t_>()) + i * chunk_size * output_batch_elems;
            hipMemcpyAsync(dst_ptr, d_temp_output[stream_idx],
                            current_chunk * out_chunk_bytes,
                            hipMemcpyDeviceToDevice, stream);
        }

        // Synchronize both streams to ensure all operations are complete
        hipStreamSynchronize(streams[0]);
        hipStreamSynchronize(streams[1]);

        // Free temporary buffers
        hipFree(d_temp_input[0]);
        hipFree(d_temp_input[1]);
        hipFree(d_temp_output[0]);
        hipFree(d_temp_output[1]);
    }));

    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Transposed 2D convolution (CUDA)",
          py::arg("x"), py::arg("weight"), py::arg("bias") = py::none(),
          py::arg("stride"), py::arg("padding"), py::arg("output_padding"),
          py::arg("groups"), py::arg("dilation") = 1);
}
