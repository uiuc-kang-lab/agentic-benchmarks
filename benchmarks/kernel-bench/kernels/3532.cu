#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Device helper: define an inline exponential function for float
__device__ inline float my_exp(float x) {
    return expf(x);
}

__device__ inline void process_element(float x, float& result) {
    result = (x > 0.0f)
        ? x 
        : 1.67326324235437728481f * (my_exp(x) - 1.0f);
    result *= 1.05070098735548049342f;
}

__global__ void selu_kernel_stride_optimized(const float* __restrict__ input,
                                      float* __restrict__ output,
                                      size_t numel) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;

    // Process elements using stride loop
    for (size_t i = idx; i < numel; i += stride) {
        float x = __ldg(&input[i]);
        float result;
        process_element(x, result);
        output[i] = result;
    }
}

torch::Tensor selu_forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");
    TORCH_CHECK(input.scalar_type() == torch::kFloat, "Input must be float32");

    auto output = torch::empty_like(input);
    const size_t numel = input.numel();
    const int threads = 1024;
    const int blocks = (numel + threads - 1) / threads;

    const float* input_ptr = input.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();
    
    selu_kernel_stride_optimized<<<blocks, threads>>>(input_ptr, output_ptr, numel);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &selu_forward, "SELU Activation Forward with Stride Optimization (CUDA)");
}