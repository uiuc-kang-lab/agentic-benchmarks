#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define TILE_DIM 16
#define BLOCK_ROWS 8
#define MATRIX_SIZE_THRESHOLD 512
#define UNROLL_FACTOR 4

__global__ void UnrolledMatmulKernel(const float* __restrict__ A,
                                    const float* __restrict__ B,
                                    float* __restrict__ C,
                                    const int M, const int K, const int N) {
    __shared__ float As[TILE_DIM][TILE_DIM];
    __shared__ float Bs[TILE_DIM][TILE_DIM];
    
    const int row = blockIdx.y * TILE_DIM + threadIdx.y;
    const int col = blockIdx.x * TILE_DIM + threadIdx.x;
    
    float sum[UNROLL_FACTOR] = {0.0f, 0.0f, 0.0f, 0.0f};
    
    // Main loop over tiles
    #pragma unroll 2
    for (int t = 0; t < (K + TILE_DIM - 1) / TILE_DIM; ++t) {
        // Load tiles into shared memory
        if (t * TILE_DIM + threadIdx.x < K && row < M) {
            As[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_DIM + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        if (t * TILE_DIM + threadIdx.y < K && col < N) {
            Bs[threadIdx.y][threadIdx.x] = B[(t * TILE_DIM + threadIdx.y) * N + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute partial products with manual unrolling
        #pragma unroll
        for (int k = 0; k < TILE_DIM; k += UNROLL_FACTOR) {
            // Manual unroll of the inner loop
            float aval[UNROLL_FACTOR];
            #pragma unroll
            for (int u = 0; u < UNROLL_FACTOR; ++u) {
                aval[u] = As[threadIdx.y][k + u];
            }
            
            #pragma unroll
            for (int u = 0; u < UNROLL_FACTOR; ++u) {
                sum[u] += aval[u] * Bs[k + u][threadIdx.x];
            }
        }
        
        __syncthreads();
    }
    
    // Accumulate final results
    float final_sum = 0.0f;
    #pragma unroll
    for (int u = 0; u < UNROLL_FACTOR; ++u) {
        final_sum += sum[u];
    }
    
    // Store result
    if (row < M && col < N) {
        C[row * N + col] = final_sum;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.is_contiguous(), "A must be contiguous");
    TORCH_CHECK(B.is_contiguous(), "B must be contiguous");

    const int M = A.size(0);
    const int K = A.size(1);
    const int N = B.size(1);

    auto C = torch::zeros({M, N}, A.options());

    if (M <= MATRIX_SIZE_THRESHOLD && N <= MATRIX_SIZE_THRESHOLD) {
        dim3 threadsPerBlock(TILE_DIM, BLOCK_ROWS);
        dim3 numBlocks(
            (N + TILE_DIM - 1) / TILE_DIM,
            (M + TILE_DIM - 1) / TILE_DIM
        );

        UnrolledMatmulKernel<<<numBlocks, threadsPerBlock>>>(
            A.data_ptr<float>(),
            B.data_ptr<float>(),
            C.data_ptr<float>(),
            M, K, N
        );
    } else {
        static hipblasHandle_t handle = nullptr;
        if (handle == nullptr) {
            hipblasCreate(&handle);
        }
        
        float alpha = 1.0f;
        float beta = 0.0f;
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                   N, M, K, &alpha,
                   B.data_ptr<float>(), N,
                   A.data_ptr<float>(), K,
                   &beta, C.data_ptr<float>(), N);
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Unrolled matrix multiplication (CUDA)");
}