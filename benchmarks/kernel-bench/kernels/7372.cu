#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

__device__ __inline__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xffffffff, val, offset);
    return val;
}

__global__ void conv2d_cuda_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int N, int C_in, int H_in, int W_in,
    int C_out, int H_out, int W_out,
    int K_h, int K_w,
    int stride_h, int stride_w,
    int padding_h, int padding_w,
    int dilation_h, int dilation_w,
    int groups
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int warp_size = 32;
    const int lane_id = threadIdx.x % warp_size;
    
    if (tid >= N * C_out * H_out * W_out) return;

    const int w_out = tid % W_out;
    int tmp = tid / W_out;
    const int h_out = tmp % H_out;
    tmp = tmp / H_out;
    const int c_out = tmp % C_out;
    const int n = tmp / C_out;

    float value = (bias != nullptr) ? bias[c_out] : 0.0f;
    const int group = c_out / (C_out / groups);
    const int c_in_start = group * (C_in / groups);

    for (int c_in = c_in_start; c_in < c_in_start + (C_in / groups); ++c_in) {
        for (int k_h = 0; k_h < K_h; ++k_h) {
            for (int k_w = 0; k_w < K_w; ++k_w) {
                const int h_in = h_out * stride_h - padding_h + k_h * dilation_h;
                const int w_in = w_out * stride_w - padding_w + k_w * dilation_w;
                
                if (h_in >= 0 && h_in < H_in && w_in >= 0 && w_in < W_in) {
                    const int input_idx = ((n * C_in + c_in) * H_in + h_in) * W_in + w_in;
                    const int weight_idx = (((c_out * (C_in / groups) + (c_in - c_in_start)) * K_h + k_h) * K_w) + k_w;
                    value += input[input_idx] * weight[weight_idx];
                }
            }
        }
    }

    // Warp-level reduction for partial sums
    value = warpReduceSum(value);
    
    if (lane_id == 0) {
        const int output_idx = ((n * C_out + c_out) * H_out + h_out) * W_out + w_out;
        output[output_idx] = value;
    }
}

torch::Tensor conv2d_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias_opt,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> dilation,
    int64_t groups
) {
    input = input.contiguous();
    weight = weight.contiguous();

    TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA");

    if (bias_opt.has_value()) {
        TORCH_CHECK(bias_opt.value().is_cuda(), "Bias tensor must be on CUDA if provided");
    }

    const int64_t N = input.size(0);
    const int64_t C_in = input.size(1);
    const int64_t H_in = input.size(2);
    const int64_t W_in = input.size(3);
    const int64_t C_out = weight.size(0);
    const int64_t K_h = weight.size(2);
    const int64_t K_w = weight.size(3);

    const int64_t H_out = (H_in + 2 * padding[0] - dilation[0] * (K_h - 1) - 1) / stride[0] + 1;
    const int64_t W_out = (W_in + 2 * padding[1] - dilation[1] * (K_w - 1) - 1) / stride[1] + 1;

    auto output = torch::zeros({N, C_out, H_out, W_out}, input.options());

    const int threads_per_block = 256;
    const int num_blocks = (N * C_out * H_out * W_out + threads_per_block - 1) / threads_per_block;

    conv2d_cuda_kernel<<<num_blocks, threads_per_block>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_opt.has_value() ? bias_opt.value().data_ptr<float>() : nullptr,
        output.data_ptr<float>(),
        N, C_in, H_in, W_in,
        C_out, H_out, W_out,
        K_h, K_w,
        stride[0], stride[1],
        padding[0], padding[1],
        dilation[0], dilation[1],
        groups
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv2d_cuda, "Warp-optimized 2D convolution (CUDA)");
}