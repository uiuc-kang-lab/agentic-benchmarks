#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__device__ __forceinline__ scalar_t warp_reduce(scalar_t val) {
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

template <typename scalar_t>
__global__ void hybrid_reduce_mean_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t outer_size,
    int64_t dim_size,
    int64_t inner_size) {
    
    constexpr int BLOCK_SIZE = 256;
    constexpr int WARP_SIZE = 32;
    __shared__ scalar_t shared_data[BLOCK_SIZE];
    
    const int tid = threadIdx.x;
    const int wid = tid / WARP_SIZE;
    const int lane = tid % WARP_SIZE;
    const int output_idx = blockIdx.x;
    
    if (output_idx >= outer_size * inner_size) return;
    
    const int outer_idx = output_idx / inner_size;
    const int inner_idx = output_idx % inner_size;
    const int input_offset = outer_idx * dim_size * inner_size + inner_idx;
    
    // First level reduction: threads cooperatively load and sum values
    scalar_t thread_sum = 0;
    #pragma unroll 4
    for (int i = tid; i < dim_size; i += BLOCK_SIZE) {
        thread_sum += input[input_offset + i * inner_size];
    }
    
    // In-warp reduction: each warp reduces its own thread_sum
    scalar_t warp_sum = warp_reduce(thread_sum);
    if (lane == 0) {
        shared_data[wid] = warp_sum;
    }
    __syncthreads();
    
    // Only the first warp reduces the warp sums from each warp
    if (wid == 0) {
        scalar_t block_sum = (lane < (BLOCK_SIZE / WARP_SIZE)) ? shared_data[lane] : 0;
        block_sum = warp_reduce(block_sum);
        if (lane == 0) {
            output[output_idx] = block_sum / static_cast<scalar_t>(dim_size);
        }
    }
}

torch::Tensor mean_reduce_cuda(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();
    
    auto sizes = input.sizes().vec();
    int64_t dim_size = sizes[dim];
    
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }
    
    sizes.erase(sizes.begin() + dim);
    auto output = torch::empty(sizes, input.options());
    
    const int BLOCK_SIZE = 256;
    const int num_blocks = outer_size * inner_size;
    
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "hybrid_reduce_mean_cuda", ([&] {
        hybrid_reduce_mean_kernel<scalar_t><<<num_blocks, BLOCK_SIZE>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            dim_size,
            inner_size
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &mean_reduce_cuda, "Mean reduction using hybrid approach (CUDA)");
}