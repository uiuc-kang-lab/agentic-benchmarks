#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

// Kernel that processes a chunk of the input starting from a given offset
template <typename scalar_t>
__global__ void softplus_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int chunk_size,
    const int offset) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < chunk_size) {
    int global_idx = idx + offset;
    const scalar_t x = input[global_idx];
    if (x > scalar_t(20.0)) {
      output[global_idx] = x;
    } else if (x < scalar_t(-20.0)) {
      output[global_idx] = exp(x);
    } else {
      output[global_idx] = log1p(exp(x));
    }
  }
}

// This function splits the input tensor into several chunks and processes each chunk
// on its own CUDA stream. The idea is to overlap memory operations (loads/stores)
// with computation by pipelining execution across streams, which can reduce overall runtime
// on large tensors, especially on hardware like the H100 with CUDA 12.2.

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
  TORCH_CHECK(input.is_cuda(), "input must be a CUDA tensor");

  auto output = torch::empty_like(input);
  const int total_elements = input.numel();

  // Define number of streams and chunk size to split the work
  const int num_streams = 4;
  const int chunk_size = (total_elements + num_streams - 1) / num_streams;

  // Create CUDA streams
  std::vector<hipStream_t> streams(num_streams);
  for (int i = 0; i < num_streams; i++) {
    hipStreamCreate(&streams[i]);
  }

  const int threads = 256;
  // Launch the kernel for each chunk on its own stream
  for (int i = 0; i < num_streams; i++) {
    int offset = i * chunk_size;
    if (offset >= total_elements) break;
    int current_chunk_size = std::min(chunk_size, total_elements - offset);
    int blocks = (current_chunk_size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "softplus_forward_cuda_stream", ([&] {
      softplus_kernel<scalar_t><<<blocks, threads, 0, streams[i]>>>(
          input.data_ptr<scalar_t>(),
          output.data_ptr<scalar_t>(),
          current_chunk_size,
          offset);
    }));
  }

  // Synchronize and destroy streams
  for (int i = 0; i < num_streams; i++) {
    hipStreamSynchronize(streams[i]);
    hipStreamDestroy(streams[i]);
  }

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &softplus_cuda_forward, "Softplus forward with stream pipeline (CUDA)");
}
