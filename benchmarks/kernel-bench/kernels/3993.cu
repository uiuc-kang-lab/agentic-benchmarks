#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define BLOCK_SIZE 256
#define SHARED_MEM_SIZE BLOCK_SIZE

__global__ void elu_kernel_shared(const float* __restrict__ x, float* __restrict__ out, float alpha, int n) {
    __shared__ float shared_data[SHARED_MEM_SIZE];
    
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;
    
    // Load data into shared memory
    if (idx < n) {
        shared_data[tid] = x[idx];
    }
    
    __syncthreads();
    
    // Process data from shared memory
    if (idx < n) {
        float val = shared_data[tid];
        out[idx] = (val > 0) ? val : alpha * (expf(val) - 1);
    }
}

torch::Tensor elu_cuda(torch::Tensor x, float alpha) {
    CHECK_INPUT(x);
    
    auto out = torch::empty_like(x);
    int n = x.numel();
    
    const int threads = BLOCK_SIZE;
    const int blocks = (n + threads - 1) / threads;
    
    elu_kernel_shared<<<blocks, threads>>>(x.data_ptr<float>(), out.data_ptr<float>(), alpha, n);
    
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &elu_cuda, "ELU activation with shared memory (CUDA)");
}