#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Define TILE_SIZE as a compile-time constant to experiment with block sizes.
#ifndef TILE_SIZE
#define TILE_SIZE 32
#endif

// CUDA kernel for computing C = A * B where A and B are lower triangular matrices using shared memory.
__global__ void triangular_mm_kernel_shared(const float* __restrict__ A,
                                             const float* __restrict__ B,
                                             float* __restrict__ C,
                                             int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.f;

    // Loop over tiles in the row dimension
    for (int tile = 0; tile < (N + TILE_SIZE - 1) / TILE_SIZE; ++tile) {
        // Load data into shared memory
        if (row < N && (tile * TILE_SIZE + threadIdx.x) < N)
            As[threadIdx.y][threadIdx.x] = A[row * N + tile * TILE_SIZE + threadIdx.x];
        else
            As[threadIdx.y][threadIdx.x] = 0.0;

        if (col < N && (tile * TILE_SIZE + threadIdx.y) < N)
            Bs[threadIdx.y][threadIdx.x] = B[(tile * TILE_SIZE + threadIdx.y) * N + col];
        else
            Bs[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        // Compute partial results
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write the result to the output matrix
    if (row < N && col < N) {
        if (row >= col) {
            C[row * N + col] = sum;
        } else {
            C[row * N + col] = 0.f;
        }
    }
}

// C++ interface exposed to PyTorch.
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    triangular_mm_kernel_shared<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized triangular matrix multiplication (CUDA) using shared memory");
}