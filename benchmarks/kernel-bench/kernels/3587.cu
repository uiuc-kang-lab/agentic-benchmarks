#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Device helper: define an inline exponential function for float and double.
template <typename scalar_t>
__device__ inline scalar_t my_exp(scalar_t x);

template <>
__device__ inline float my_exp<float>(float x) {
    return expf(x);
}

template <>
__device__ inline double my_exp<double>(double x) {
    return exp(x);
}

// Optimized SELU kernel using loop striding. Each thread processes multiple elements,
// which reduces kernel launch overhead and improves occupancy without needing atomic operations.
template <typename scalar_t>
__global__ void selu_kernel_strided(const scalar_t* __restrict__ input,
                                    scalar_t* __restrict__ output,
                                    size_t numel) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    const scalar_t alpha = static_cast<scalar_t>(1.67326324235437728481);
    const scalar_t lambda = static_cast<scalar_t>(1.05070098735548049342);

    for (size_t i = idx; i < numel; i += stride) {
        scalar_t x = input[i];
        scalar_t result = (x > static_cast<scalar_t>(0))
                              ? x
                              : alpha * (my_exp(x) - static_cast<scalar_t>(1));
        output[i] = lambda * result;
    }
}

// Host function launching the optimized SELU kernel.
// Since the operation is entirely element-wise, there is no need for atomic operations.
// This minimizes global memory contention and helps reduce runtime.

torch::Tensor selu_forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    auto output = torch::empty_like(input);
    size_t numel = input.numel();
    const int threads = 1024;
    const int blocks = (numel + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "selu_forward_cuda_strided", ([&] {
        const scalar_t* input_ptr = input.data_ptr<scalar_t>();
        scalar_t* output_ptr = output.data_ptr<scalar_t>();
        selu_kernel_strided<scalar_t><<<blocks, threads>>>(input_ptr, output_ptr, numel);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &selu_forward, "SELU Activation Forward (CUDA Strided Kernel)");
}
