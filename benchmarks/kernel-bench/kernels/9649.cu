#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

// This kernel combines improved occupancy and minimized warp divergence.
// It uses 512 threads per block and precomputes valid index ranges for the convolution window.

template <typename scalar_t>
__global__ void optimizedDepthwiseConv2DKernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ w,
    const scalar_t* __restrict__ b,
    scalar_t* __restrict__ out,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int kernel_size,
    const int out_height,
    const int out_width,
    const int stride,
    const int padding) {

    const int w_out_idx = threadIdx.x + blockIdx.x * blockDim.x;
    const int h_out_idx = threadIdx.y + blockIdx.y * blockDim.y;
    const int c = blockIdx.z % in_channels;
    const int n = blockIdx.z / in_channels;

    if (w_out_idx >= out_width || h_out_idx >= out_height || n >= batch_size) {
        return;
    }

    scalar_t value = 0;

    // Precompute valid kernel range to avoid divergence
    const int h_in_start = h_out_idx * stride - padding;
    const int w_in_start = w_out_idx * stride - padding;

    const int kh_start = max(0, -h_in_start);
    const int kh_end = min(kernel_size, in_height - h_in_start);
    const int kw_start = max(0, -w_in_start);
    const int kw_end = min(kernel_size, in_width - w_in_start);

    // Pre-calculate base indices to reduce arithmetic in inner loops
    const int batch_channel_offset = (n * in_channels + c) * in_height;
    const int kernel_channel_offset = c * kernel_size * kernel_size;

    #pragma unroll
    for (int kh = kh_start; kh < kh_end; kh++) {
        const int h_in = h_in_start + kh;
        const int x_h_offset = (batch_channel_offset + h_in) * in_width;
        const int w_h_offset = (kernel_channel_offset + kh * kernel_size);
        
        #pragma unroll
        for (int kw = kw_start; kw < kw_end; kw++) {
            const int w_in = w_in_start + kw;
            const int x_index = x_h_offset + w_in;
            const int w_index = w_h_offset + kw;
            value += __ldg(&x[x_index]) * __ldg(&w[w_index]);
        }
    }

    value += b[c];

    const int out_idx = ((n * in_channels + c) * out_height + h_out_idx) * out_width + w_out_idx;
    out[out_idx] = value;
}

// Forward implementation that configures the kernel launch parameters using a block size of 512 threads
// (32 threads in x and 16 in y) to improve occupancy and reduce latency on the H100 GPU.

torch::Tensor forward_impl(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int groups) {

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);
    const int kernel_size = weight.size(2);
    const int out_height = (in_height + 2 * padding - kernel_size) / stride + 1;
    const int out_width  = (in_width + 2 * padding - kernel_size) / stride + 1;

    auto out = torch::empty({batch_size, in_channels, out_height, out_width}, x.options());

    // Block configuration: 32x16 gives 512 threads per block, which can better hide memory latency.
    dim3 threads(32, 16);
    // Calculate grid dimensions using ceiling division to exactly cover the output dimensions
    dim3 blocks(
        (out_width + threads.x - 1) / threads.x,
        (out_height + threads.y - 1) / threads.y,
        batch_size * in_channels
    );
    
    // Ensure we don't launch more blocks than necessary for the problem size
    blocks.x = min(blocks.x, (unsigned int)((out_width + threads.x - 1) / threads.x));
    blocks.y = min(blocks.y, (unsigned int)((out_height + threads.y - 1) / threads.y));

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "depthwise_conv2d_forward", ([&] {
        optimizedDepthwiseConv2DKernel<scalar_t><<<blocks, threads>>>(
            x.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            bias.data_ptr<scalar_t>(),
            out.data_ptr<scalar_t>(),
            batch_size, in_channels, in_height, in_width,
            kernel_size, out_height, out_width,
            stride, padding
        );
    }));

    return out;
}

namespace py = pybind11;

// Wrap forward_impl to handle optional bias input
torch::Tensor forward_wrap(
    torch::Tensor x,
    torch::Tensor weight,
    py::object bias_obj,
    int stride,
    int padding,
    int groups) {
    torch::Tensor bias;
    if (bias_obj.is_none()) {
        bias = torch::zeros({x.size(1)}, x.options());
    } else {
        bias = bias_obj.cast<torch::Tensor>();
    }
    return forward_impl(x, weight, bias, stride, padding, groups);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        &forward_wrap,
        "Optimized depthwise conv2d forward",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = 1,
        py::arg("padding") = 0,
        py::arg("groups") = 1
    );
}
