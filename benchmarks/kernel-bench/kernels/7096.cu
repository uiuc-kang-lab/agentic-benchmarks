#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <c10/cuda/CUDAStream.h>

// CUDA kernel that performs min reduction over a specified dimension with loop unrolling by factor 8
// The input is logically viewed as [outer, r, inner] and the reduction is performed over dimension r.

template <typename scalar_t>
__global__ void min_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int outer,
    const int r,
    const int inner) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = outer * inner;
  if (idx >= total) return;

  int outer_idx = idx / inner;
  int inner_idx = idx % inner;
  int base_offset = outer_idx * r * inner; // starting position for this reduction

  // Initialize min_val with the first element
  scalar_t min_val = input[base_offset + inner_idx];

  // Unroll the loop in chunks of 8 starting from j = 1
  int j_stop = ((r - 1) / 8) * 8 + 1;  // Process in multiples of 8
  #pragma unroll
  for (int j = 1; j < j_stop; j += 8) {
    int idx0 = base_offset + (j + 0) * inner + inner_idx;
    int idx1 = base_offset + (j + 1) * inner + inner_idx;
    int idx2 = base_offset + (j + 2) * inner + inner_idx;
    int idx3 = base_offset + (j + 3) * inner + inner_idx;
    int idx4 = base_offset + (j + 4) * inner + inner_idx;
    int idx5 = base_offset + (j + 5) * inner + inner_idx;
    int idx6 = base_offset + (j + 6) * inner + inner_idx;
    int idx7 = base_offset + (j + 7) * inner + inner_idx;

    min_val = min(min_val, input[idx0]);
    min_val = min(min_val, input[idx1]);
    min_val = min(min_val, input[idx2]);
    min_val = min(min_val, input[idx3]);
    min_val = min(min_val, input[idx4]);
    min_val = min(min_val, input[idx5]);
    min_val = min(min_val, input[idx6]);
    min_val = min(min_val, input[idx7]);
  }

  // Process any remaining elements
  for (int j = j_stop; j < r; j++) {
    int index = base_offset + j * inner + inner_idx;
    min_val = min(min_val, input[index]);
  }

  output[idx] = min_val;
}

// The forward function wraps the kernel launch and handles tensor shape rearrangement

torch::Tensor forward(torch::Tensor input, int64_t dim) {
  TORCH_CHECK(input.is_cuda(), "input must be a CUDA tensor");
  if (!input.is_contiguous()) {
    input = input.contiguous();
  }

  int ndim = input.dim();
  TORCH_CHECK(dim >= 0 && dim < ndim, "dim out of range");

  // Compute the sizes for outer, reduction dimension (r), and inner dimensions
  int outer = 1;
  for (int i = 0; i < dim; i++) {
    outer *= input.size(i);
  }
  int r = input.size(dim);
  int inner = 1;
  for (int i = dim + 1; i < ndim; i++) {
    inner *= input.size(i);
  }

  // Form the output shape by removing the reduced dimension
  std::vector<int64_t> output_shape;
  for (int i = 0; i < ndim; i++) {
    if (i != dim) {
      output_shape.push_back(input.size(i));
    }
  }

  auto output = torch::empty(output_shape, input.options());

  int total = outer * inner;
  const int threads = 256;
  const int blocks = (total + threads - 1) / threads;

  AT_DISPATCH_ALL_TYPES(input.scalar_type(), "min_reduce_cuda", ([&] {
    min_reduce_kernel<scalar_t><<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream().stream()>>>(
        input.data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>(),
        outer,
        r,
        inner);
  }));

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "Min reduction over a specified dimension (CUDA)");
}
