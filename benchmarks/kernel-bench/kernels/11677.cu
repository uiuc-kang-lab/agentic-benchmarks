#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void optimized_kl_div_thread_block_indexing_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    // Use 1D grid and 1D block for simplicity
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Shared memory for partial sums
    extern __shared__ float partial_sums[];
    
    float sum = 0.0f;
    
    // Iterate over the data with stride
    for (int i = idx; i < n; i += stride) {
        float log_pred = log_predictions[i];
        float target = targets[i];
        sum += expf(log_pred) - target * log_pred;
    }
    
    // Store partial sum in shared memory
    partial_sums[threadIdx.x] = sum;
    __syncthreads();
    
    // Reduce within block using a single warp
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            partial_sums[threadIdx.x] += partial_sums[threadIdx.x + offset];
        }
        __syncthreads();
    }
    
    // Write block's result to global memory
    if (threadIdx.x == 0) {
        atomicAdd(output, partial_sums[0]);
    }
}

torch::Tensor optimized_kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;
    const int shared_mem = threads * sizeof(float);
    
    optimized_kl_div_thread_block_indexing_kernel<<<blocks, threads, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &optimized_kl_div_cuda_forward, "Optimized KL divergence forward with thread block indexing (CUDA)");
}