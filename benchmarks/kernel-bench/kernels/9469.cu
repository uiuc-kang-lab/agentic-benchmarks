#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// Device function to decode a linear index into (w_out, h_out, o, b)
__device__ inline void decode_index(int index, int out_width, int out_height, int out_channels,
                                    int &w_out, int &h_out, int &o, int &b) {
    w_out = index % out_width;
    int temp = index / out_width;
    h_out = temp % out_height;
    temp /= out_height;
    o = temp % out_channels;
    b = temp / out_channels;
}

// Device function to compute the contribution from one input channel
__device__ inline float compute_channel_contribution(
    const float* input,
    const float* weight,
    int b,
    int c,
    int o,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_h,
    int out_w,
    int pad,
    int dilation,
    int stride) {

    float sum = 0.0f;
    for (int p = 0; p < kernel_size; ++p) {
        int h_unscaled = out_h + pad - p * dilation;
        if (h_unscaled % stride != 0)
            continue;
        int h_in = h_unscaled / stride;
        if (h_in < 0 || h_in >= in_height)
            continue;
        for (int q = 0; q < kernel_size; ++q) {
            int w_unscaled = out_w + pad - q * dilation;
            if (w_unscaled % stride != 0)
                continue;
            int w_in = w_unscaled / stride;
            if (w_in < 0 || w_in >= in_width)
                continue;
            int input_idx = ((b * in_channels + c) * in_height + h_in) * in_width + w_in;
            int weight_idx = ((c * out_channels + o) * kernel_size + p) * kernel_size + q;
            sum += input[input_idx] * weight[weight_idx];
        }
    }
    return sum;
}

// CUDA kernel refactored with modular device functions for clarity
__global__ void conv_transpose2d_forward_kernel_modular(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch_size * out_channels * out_height * out_width;
    if (index >= total)
        return;

    int w_out, h_out, o, b;
    decode_index(index, out_width, out_height, out_channels, w_out, h_out, o, b);

    float out_val = bias[o];
    for (int c = 0; c < in_channels; ++c) {
        out_val += compute_channel_contribution(input, weight, b, c, o,
                                                  in_channels, out_channels,
                                                  in_height, in_width,
                                                  kernel_size, h_out, w_out,
                                                  padding, dilation, stride);
    }
    int output_idx = ((b * out_channels + o) * out_height + h_out) * out_width + w_out;
    output[output_idx] = out_val;
}

// CUDA launcher function
torch::Tensor conv_transpose2d_forward_cuda_modular(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {

    int batch_size = input.size(0);
    int in_channels = input.size(1);
    int in_height = input.size(2);
    int in_width = input.size(3);

    int out_channels = weight.size(1);
    int kernel_size = weight.size(2);  // assume square kernel

    int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
    int out_width  = (in_width - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;

    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());

    int total_threads = batch_size * out_channels * out_height * out_width;
    int threads = 1024;
    int blocks = (total_threads + threads - 1) / threads;

    conv_transpose2d_forward_kernel_modular<<<blocks, threads>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        out_channels,
        in_height,
        in_width,
        kernel_size,
        out_height,
        out_width,
        stride,
        padding,
        dilation);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in conv_transpose2d_forward_kernel_modular: %s\n", hipGetErrorString(err));
    }

    return output;
}

// Wrapper to handle the case when bias is None
torch::Tensor conv_transpose2d_forward_wrapper_modular(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,
    int stride,
    int padding,
    int dilation) {

    int out_channels = weight.size(1);
    torch::Tensor bias;
    if (bias_obj.is(pybind11::none())) {
        bias = torch::zeros({out_channels}, weight.options());
    } else {
        bias = bias_obj.cast<torch::Tensor>();
    }

    return conv_transpose2d_forward_cuda_modular(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward_wrapper_modular,
          "ConvTranspose2d forward modular (CUDA)",
          pybind11::arg("input"),
          pybind11::arg("weight"),
          pybind11::arg("bias"),
          pybind11::arg("stride"),
          pybind11::arg("padding"),
          pybind11::arg("dilation"));
}
