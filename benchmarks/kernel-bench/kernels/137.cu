#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Static cuBLAS handle
static hipblasHandle_t handle = nullptr;

void matrix_multiply_cuda(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);

    const int M = A.size(0);
    const int K = A.size(1);
    const int N = B.size(1);

    const float* d_A = A.data_ptr<float>();
    const float* d_B = B.data_ptr<float>();
    float* d_C = C.data_ptr<float>();

    // Initialize cuBLAS handle if needed
    if (handle == nullptr) {
        hipblasCreate(&handle);
        // Enable Tensor Cores
        hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    }

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Direct cuBLAS call optimized for row-major input
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, M, K,
                &alpha,
                d_B, N,  // B's leading dimension
                d_A, K,  // A's leading dimension
                &beta,
                d_C, N); // C's leading dimension
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    const int M = A.size(0);
    const int N = B.size(1);

    auto options = torch::TensorOptions()
                       .dtype(A.dtype())
                       .device(A.device())
                       .requires_grad(false);
    
    torch::Tensor C = torch::empty({M, N}, options);
    matrix_multiply_cuda(A, B, C);
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Direct cuBLAS matrix multiplication (CUDA)");
}