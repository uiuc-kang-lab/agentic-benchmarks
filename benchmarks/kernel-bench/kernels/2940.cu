#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void tanh_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process multiple elements per thread to reduce overall thread synchronization needs
    for (int i = idx; i < size; i += blockDim.x * gridDim.x) {
        output[i] = tanhf(input[i]);
    }
}

torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    
    const int threads = 256;
    const int blocks = min(65535, (input.numel() + threads - 1) / threads);
    
    AT_DISPATCH_FLOATING_TYPES(input.type(), "tanh_kernel", ([&] {
        tanh_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            input.numel()
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Tanh forward (CUDA)");
}