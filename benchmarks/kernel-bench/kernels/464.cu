#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

__device__ float multiplyElement(float a, float s) {
    return a * s;
}

__global__ void multiplyKernel(const float* __restrict__ A,
                               float* __restrict__ C,
                               float s,
                               int64_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        C[idx] = multiplyElement(A[idx], s);
    }
}

torch::Tensor forward(torch::Tensor A, float s)
{
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor.");
    TORCH_CHECK(A.scalar_type() == torch::kFloat, "Input tensor A must be of type float.");

    auto C = torch::empty_like(A);
    int64_t size = A.numel();
    const int threads = 256;
    const int blocks = (size + threads - 1) / threads;

    multiplyKernel<<<blocks, threads>>>(A.data_ptr<float>(),
                                       C.data_ptr<float>(),
                                       s,
                                       size);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix-scalar multiplication kernel");
}