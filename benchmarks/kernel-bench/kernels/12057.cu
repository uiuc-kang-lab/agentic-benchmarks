#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void hinge_loss_stride_kernel(const float* __restrict__ predictions, 
                                         const float* __restrict__ targets, 
                                         float* output, 
                                         int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = idx; i < n; i += stride) {
        float pred = __ldg(&predictions[i]);
        float targ = __ldg(&targets[i]);
        output[i] = fmaxf(0.0f, 1.0f - pred * targ);
    }
}

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    CHECK_INPUT(predictions);
    CHECK_INPUT(targets);

    int n = predictions.numel();
    torch::Tensor output = torch::empty_like(predictions);

    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    hinge_loss_stride_kernel<<<blocks, threads>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    // Compute the mean of the output tensor
    return torch::mean(output);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Stride Hinge Loss Forward");
}
