#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void relu_kernel_strided(
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ input,
    const int64_t size) {
    
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    // Each thread processes multiple elements with stride
    for (int idx = tid; idx < size; idx += stride) {
        const scalar_t val = input[idx];
        output[idx] = val > 0 ? val : 0;
    }
}

torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    
    const int threads = 256;
    const int blocks = std::min(65535, (int)((input.numel() + threads - 1) / threads));
    
    AT_DISPATCH_FLOATING_TYPES(input.type(), "relu_kernel_strided", ([&] {
        relu_kernel_strided<scalar_t><<<blocks, threads>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            input.numel()
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ReLU forward strided (CUDA)");
}