#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void elu_kernel_coalesced(const float* __restrict__ x, float* __restrict__ out, float alpha, int n) __attribute__((optimize("O3"))) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = (x[idx] > 0) ? x[idx] : alpha * (expf(x[idx]) - 1);
    }
}

torch::Tensor elu_cuda_coalesced(torch::Tensor x, float alpha) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int n = x.numel();

    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;

    elu_kernel_coalesced<<<blocks, threads>>>(x.data_ptr<float>(), out.data_ptr<float>(), alpha, n);

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &elu_cuda_coalesced, "ELU activation with memory coalescing (CUDA)");
}