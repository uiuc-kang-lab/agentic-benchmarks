#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

#define WARP_SIZE 32

// CUDA kernel implementing L1 normalization with balanced workload distribution
__global__ void l1_norm_forward_kernel_balanced(const float* __restrict__ x,
                                                 float* __restrict__ out,
                                                 int N,
                                                 int D) {
  // Calculate the global thread index and corresponding row
  int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row = global_idx / D;
  int col = global_idx % D;

  if (row >= N) return;  // Ensure row is within bounds

  // Each thread calculates the absolute value for its element
  float thread_sum = (col < D) ? fabsf(x[row * D + col]) : 0.0f;

  // Warp-level reduction using shuffle intrinsics
  for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
    thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
  }

  // Shared memory reduction across warps
  extern __shared__ float sdata[];
  int warp_id = threadIdx.x / WARP_SIZE;
  if (threadIdx.x % WARP_SIZE == 0) {
    sdata[warp_id] = thread_sum;
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    float total_sum = 0.0f;
    int num_warps = (blockDim.x + WARP_SIZE - 1) / WARP_SIZE;
    for (int i = 0; i < num_warps; i++) {
      total_sum += sdata[i];
    }
    if (total_sum == 0.0f) {
      total_sum = 1e-12f;
    }
    sdata[0] = total_sum;
  }
  __syncthreads();
  float total_sum = sdata[0];

  // Normalize the row elements
  if (col < D) {
    out[row * D + col] = x[row * D + col] / total_sum;
  }
}

// Host function to launch the CUDA kernel

torch::Tensor forward(torch::Tensor x) {
  TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA.");
  TORCH_CHECK(x.dim() == 2, "Expected 2D tensor.");
  x = x.contiguous();

  auto out = torch::empty_like(x);
  int N = x.size(0);
  int D = x.size(1);

  // Determine the number of threads per block
  int threads = 256;  // Choose a balanced number of threads
  int blocks = (N * D + threads - 1) / threads;
  int num_warps = (threads + WARP_SIZE - 1) / WARP_SIZE;
  int shared_mem_size = num_warps * sizeof(float);

  l1_norm_forward_kernel_balanced<<<blocks, threads, shared_mem_size>>>(
    x.data_ptr<float>(),
    out.data_ptr<float>(),
    N,
    D
  );

  return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "L1 Normalization forward pass with balanced workload (CUDA)");
}
