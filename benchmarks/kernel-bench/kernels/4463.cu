#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/cuda/HIPContext.h>
#include <vector>

// Declare constant memory for weight and bias
__constant__ float c_weight[1024];  // Assuming max channels <= 1024
__constant__ float c_bias[1024];

// Kernel to compute per-group mean and variance
template <typename scalar_t>
__global__ void compute_stats_kernel(
    const scalar_t* __restrict__ x,
    const int N,
    const int C,
    const int spatial,
    const int channels_per_group,
    const int num_groups,
    scalar_t* __restrict__ mean,
    scalar_t* __restrict__ var) {

  const int idx = blockIdx.x;
  const int n = idx / num_groups;
  const int g = idx % num_groups;
  
  const int group_offset = n * C * spatial + g * channels_per_group * spatial;
  const int group_elems = channels_per_group * spatial;

  scalar_t sum = 0;
  scalar_t sum_sq = 0;
  for (int i = threadIdx.x; i < group_elems; i += blockDim.x) {
    const int c = i / spatial;
    const int j = i % spatial;
    const scalar_t val = x[group_offset + c * spatial + j];
    sum += val;
    sum_sq += val * val;
  }

  extern __shared__ char smem[];
  scalar_t* s_sum = reinterpret_cast<scalar_t*>(smem);
  scalar_t* s_sum_sq = s_sum + blockDim.x;
  s_sum[threadIdx.x] = sum;
  s_sum_sq[threadIdx.x] = sum_sq;
  __syncthreads();

  for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
    if (threadIdx.x < stride) {
      s_sum[threadIdx.x] += s_sum[threadIdx.x + stride];
      s_sum_sq[threadIdx.x] += s_sum_sq[threadIdx.x + stride];
    }
    __syncthreads();
  }

  if (threadIdx.x == 0) {
    const scalar_t group_mean = s_sum[0] / group_elems;
    const scalar_t group_var = s_sum_sq[0] / group_elems - group_mean * group_mean;
    const int out_index = n * num_groups + g;
    mean[out_index] = group_mean;
    var[out_index] = group_var;
  }
}

// Modified kernel using constant memory for weight and bias
template <typename scalar_t>
__global__ void group_norm_forward_kernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ mean,
    const scalar_t* __restrict__ var,
    const int N,
    const int C,
    const int spatial,
    const int channels_per_group,
    const int num_groups,
    const scalar_t eps,
    scalar_t* __restrict__ y) {

  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int total = N * C * spatial;
  if (index >= total) return;

  const int j = index % spatial;
  const int temp = index / spatial;
  const int c = temp % C;
  const int n = temp / C;

  const int g = c / channels_per_group;
  const int stats_index = n * num_groups + g;
  const scalar_t m = mean[stats_index];
  const scalar_t v = var[stats_index];
  const scalar_t inv_std = rsqrt(v + eps);
  const scalar_t x_val = x[index];
  // Use constant memory for weight and bias
  y[index] = (x_val - m) * inv_std * c_weight[c] + c_bias[c];
}

torch::Tensor group_norm_forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    int64_t num_groups,
    double eps) {

  const int N = x.size(0);
  const int C = x.size(1);

  int spatial = 1;
  for (int i = 2; i < x.dim(); i++) {
    spatial *= x.size(i);
  }
  const int channels_per_group = C / num_groups;

  auto y = torch::empty_like(x);
  auto options = torch::TensorOptions().device(x.device()).dtype(x.dtype());
  auto mean = torch::empty({N, num_groups}, options);
  auto var = torch::empty({N, num_groups}, options);

  // Copy weight and bias to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_weight), weight.data_ptr<float>(), C * sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(c_bias), bias.data_ptr<float>(), C * sizeof(float));

  const int total_groups = N * num_groups;
  const int group_elems = channels_per_group * spatial;
  const int threads_stats = (group_elems < 256 ? group_elems : 256);
  const dim3 blocks_stats(total_groups);

  const int total_elements = N * C * spatial;
  const int threads_norm = 256;
  const dim3 blocks_norm((total_elements + threads_norm - 1) / threads_norm);

  hipStream_t stream = c10::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "group_norm_forward_cuda", ([&] {
    const size_t shared_mem_size = threads_stats * 2 * sizeof(scalar_t);
    compute_stats_kernel<scalar_t><<<blocks_stats, threads_stats, shared_mem_size, stream>>>(
        x.data_ptr<scalar_t>(),
        N,
        C,
        spatial,
        channels_per_group,
        num_groups,
        mean.data_ptr<scalar_t>(),
        var.data_ptr<scalar_t>());

    group_norm_forward_kernel<scalar_t><<<blocks_norm, threads_norm, 0, stream>>>(
        x.data_ptr<scalar_t>(),
        mean.data_ptr<scalar_t>(),
        var.data_ptr<scalar_t>(),
        N,
        C,
        spatial,
        channels_per_group,
        num_groups,
        static_cast<scalar_t>(eps),
        y.data_ptr<scalar_t>());
  }));

  return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &group_norm_forward, "Group Normalization forward (CUDA)");
}