#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void softsign_kernel_stride(const float* __restrict__ x, float* __restrict__ out, int num_elements) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    // Stride loop over data
    for (int i = tid; i < num_elements; i += stride) {
        float val = x[i];
        out[i] = val / (1.0f + fabsf(val));
    }
}

torch::Tensor forward(torch::Tensor x) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int num_elements = x.numel();
    int threads = 256;  // Updated for better warp occupancy
    const int max_blocks = 2048;  // Avoid oversubscription
    int blocks = (num_elements + threads - 1) / threads; blocks = min(blocks, max_blocks);

    softsign_kernel_stride<<<blocks, threads>>>(
        x.data_ptr<float>(), out.data_ptr<float>(), num_elements
    );

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Softsign activation with stride optimization (CUDA)");
}
