#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void diag_matmul_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    const int64_t N,
    const int64_t M
) {
    extern __shared__ float shared_A[];
    const int row = blockIdx.x;
    if (row < N) {
        // Load diagonal element into shared memory
        if (threadIdx.x == 0) {
            shared_A[0] = A[row];
        }
        __syncthreads();
        float a_val = shared_A[0];

        // Each thread processes a column
        for (int col = threadIdx.x; col < M; col += blockDim.x) {
            int idx = row * M + col;
            C[idx] = a_val * B[idx];
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.dim() == 1, "A must be a 1D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == B.size(0),
                "Dimension mismatch: A.size(0) must match B.size(0)");

    A = A.contiguous();
    B = B.contiguous();

    int64_t N = A.size(0);
    int64_t M = B.size(1);

    auto C = torch::empty({N, M}, B.options());

    const int threads = 256;
    const int blocks = N;
    size_t shared_mem_size = sizeof(float);

    diag_matmul_kernel<<<blocks, threads, shared_mem_size>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N,
        M
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Shared memory optimized diagonal matrix multiplication");
}
