#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32
#define WARP_SIZE 32
#define TILE_M 8
#define TILE_N 4

__global__ void matmul_hierarchical_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    int warp_id = tid / WARP_SIZE;
    int lane = tid % WARP_SIZE;

    int warp_row = warp_id / TILE_N;
    int warp_col = warp_id % TILE_N;

    int block_row = blockIdx.y * TILE_SIZE;
    int block_col = blockIdx.x * TILE_SIZE;

    int m = block_row + warp_row;  // Row index of warp within block
    int n = block_col + warp_col;  // Column index of warp within block

    float sum = 0.0f;

    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        int k_offset = t * TILE_SIZE;

        if ((block_row + threadIdx.y) < M && (k_offset + threadIdx.x) < K) {
            As[threadIdx.y][threadIdx.x] = __ldg(&A[(block_row + threadIdx.y) * K + k_offset + threadIdx.x]);
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0;
        }

        if ((block_col + threadIdx.y) < N && (k_offset + threadIdx.x) < K) {
            Bs[threadIdx.y][threadIdx.x] = __ldg(&B[(block_col + threadIdx.y) * K + k_offset + threadIdx.x]);
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads();

        for (int k = lane; k < TILE_SIZE; k += WARP_SIZE) {
            sum += As[warp_row * TILE_M + threadIdx.y][k] * Bs[k][warp_col * TILE_N + threadIdx.x];
        }

        __syncthreads();
    }

    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    if (lane == 0 && m < M && n < N) {
        C[m * N + n] = sum;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.dim() == 2, "A must be 2D");
    TORCH_CHECK(B.dim() == 2, "B must be 2D");
    TORCH_CHECK(A.size(1) == B.size(1), "A and B must have same K dimension");
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be on CUDA");
    TORCH_CHECK(A.is_contiguous() && B.is_contiguous(), "Inputs must be contiguous");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(0);

    auto C = torch::empty({M, N}, A.options());

    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);

    matmul_hierarchical_kernel<<<grid, block>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, N, K
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication optimized with hierarchical tiling and warp-level reduction (CUDA)");
}
