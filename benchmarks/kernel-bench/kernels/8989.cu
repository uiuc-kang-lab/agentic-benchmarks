#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Hybrid kernel that combines shared memory for weights and loop unrolling
__global__ void conv1d_kernel_hybrid(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int in_channels,
    int in_size,
    int out_size,
    int kernel_size,
    int stride,
    int dilation
) {
    int b = blockIdx.x;
    int oc = blockIdx.y;
    
    // Shared memory for weights
    extern __shared__ float sweight[];
    int filter_size = in_channels * kernel_size;
    
    // Cooperatively load weights into shared memory
    #pragma unroll 4
    for (int i = threadIdx.x; i < filter_size; i += blockDim.x) {
        sweight[i] = weight[oc * filter_size + i];
    }
    __syncthreads();
    
    float bias_val = (bias != nullptr) ? bias[oc] : 0.0f;
    
    // Grid-stride loop over output positions
    for (int o = threadIdx.x; o < out_size; o += blockDim.x) {
        float sum = 0.0f;
        
        // Loop over input channels
        for (int ic = 0; ic < in_channels; ++ic) {
            // Unroll kernel loop for better instruction-level parallelism
            #pragma unroll
            for (int k = 0; k < kernel_size; ++k) {
                int input_pos = o * stride + k * dilation;
                if (input_pos < in_size) {
                    int x_idx = b * (in_channels * in_size) + ic * in_size + input_pos;
                    sum += x[x_idx] * sweight[ic * kernel_size + k];
                }
            }
        }
        
        sum += bias_val;
        int out_idx = b * (gridDim.y * out_size) + oc * out_size + o;
        output[out_idx] = sum;
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int dilation
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(weight.dim() == 3, "weight must be 3D");
    TORCH_CHECK(weight.size(1) == x.size(1), "Input channels mismatch");

    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        TORCH_CHECK(bias->dim() == 1, "bias must be 1D");
        TORCH_CHECK(bias->size(0) == weight.size(0), "Bias size mismatch");
    }

    int B = x.size(0);
    int in_channels = x.size(1);
    int in_size = x.size(2);
    int out_channels = weight.size(0);
    int kernel_size = weight.size(2);

    int out_size = (in_size - dilation * (kernel_size - 1) - 1) / stride + 1;
    TORCH_CHECK(out_size > 0, "Invalid output size");

    auto output = torch::empty({B, out_channels, out_size}, x.options());
    if (output.numel() == 0) return output;

    const float* x_data = x.data_ptr<float>();
    const float* weight_data = weight.data_ptr<float>();
    const float* bias_data = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float* output_data = output.data_ptr<float>();

    dim3 blocks(B, out_channels);
    int threads = 256;
    int shared_mem_size = in_channels * kernel_size * sizeof(float);

    conv1d_kernel_hybrid<<<blocks, threads, shared_mem_size>>>(
        x_data,
        weight_data,
        bias_data,
        output_data,
        in_channels,
        in_size,
        out_size,
        kernel_size,
        stride,
        dilation
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel launch error: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "1D convolution forward (CUDA) with hybrid optimization");
}