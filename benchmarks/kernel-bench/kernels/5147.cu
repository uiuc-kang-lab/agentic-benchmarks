#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ATen/AccumulateType.h>

#define NUM_STREAMS 4
#define CHUNK_SIZE 1024

template <typename scalar_t>
__global__ void layernorm_forward_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    const float eps,
    scalar_t* __restrict__ output,
    const int normalized_size,
    const int chunk_offset) {

    int instance_idx = blockIdx.x;
    int tid = threadIdx.x;
    
    const scalar_t* in_ptr = input + (instance_idx + chunk_offset) * normalized_size;
    scalar_t* out_ptr = output + (instance_idx + chunk_offset) * normalized_size;

    using accscalar_t = at::acc_type<scalar_t, true>;

    extern __shared__ char smem[];
    accscalar_t* s_sum = reinterpret_cast<accscalar_t*>(smem);
    accscalar_t* s_sum_sq = s_sum + blockDim.x;

    accscalar_t local_sum = 0;
    accscalar_t local_sum_sq = 0;
    
    #pragma unroll
    for (int i = tid; i < normalized_size; i += blockDim.x) {
        accscalar_t val = static_cast<accscalar_t>(in_ptr[i]);
        local_sum += val;
        local_sum_sq += val * val;
    }
    
    s_sum[tid] = local_sum;
    s_sum_sq[tid] = local_sum_sq;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            s_sum[tid] += s_sum[tid + stride];
            s_sum_sq[tid] += s_sum_sq[tid + stride];
        }
        __syncthreads();
    }

    __shared__ accscalar_t mean;
    __shared__ accscalar_t inv_std;
    if (tid == 0) {
        mean = s_sum[0] / static_cast<accscalar_t>(normalized_size);
        accscalar_t var = s_sum_sq[0] / static_cast<accscalar_t>(normalized_size) - mean * mean;
        inv_std = static_cast<accscalar_t>(1) / sqrt(var + static_cast<accscalar_t>(eps));
    }
    __syncthreads();

    #pragma unroll
    for (int i = tid; i < normalized_size; i += blockDim.x) {
        accscalar_t val = static_cast<accscalar_t>(in_ptr[i]);
        accscalar_t norm_val = (val - mean) * inv_std;
        out_ptr[i] = static_cast<scalar_t>(norm_val * static_cast<accscalar_t>(weight[i]) +
                                         static_cast<accscalar_t>(bias[i]));
    }
}

torch::Tensor layernorm_forward(torch::Tensor x, torch::Tensor weight, torch::Tensor bias, double eps = 1e-5) {
    auto output = torch::empty_like(x);
    
    int normalized_size = weight.numel();
    int outer_size = x.numel() / normalized_size;
    
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    int threads = (normalized_size < 1024) ? normalized_size : 1024;
    int chunks_per_stream = (outer_size + NUM_STREAMS - 1) / NUM_STREAMS;
    
    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "layernorm_forward_cuda", ([&] {
        using accscalar_t = at::acc_type<scalar_t, true>;
        int shared_size = threads * 2 * sizeof(accscalar_t);
        
        for (int stream_idx = 0; stream_idx < NUM_STREAMS; stream_idx++) {
            int chunk_offset = stream_idx * chunks_per_stream;
            int chunk_size = std::min(chunks_per_stream, outer_size - chunk_offset);
            
            if (chunk_size <= 0) continue;
            
            layernorm_forward_kernel<scalar_t><<<chunk_size, threads, shared_size, streams[stream_idx]>>>(
                x.data_ptr<scalar_t>(),
                weight.data_ptr<scalar_t>(),
                bias.data_ptr<scalar_t>(),
                static_cast<float>(eps),
                output.data_ptr<scalar_t>(),
                normalized_size,
                chunk_offset);
        }
    }));

    // Synchronize all streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &layernorm_forward, "LayerNorm forward (CUDA)",
          py::arg("x"), py::arg("weight"), py::arg("bias"), py::arg("eps") = 1e-5);
}