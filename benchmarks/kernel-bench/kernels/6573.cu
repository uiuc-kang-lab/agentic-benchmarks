#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// This kernel implements max reduction over a specified dimension using loop unrolling
// with #pragma unroll to reduce loop overhead for improved performance.

template <typename scalar_t>
__global__ void max_reduce_unroll_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t outer_size,
    const int64_t dim_size,
    const int64_t inner_size
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = outer_size * inner_size;
    if (idx >= total_elements) return;

    int outer_idx = idx / inner_size;
    int inner_idx = idx % inner_size;
    int64_t base = outer_idx * dim_size * inner_size + inner_idx;

    // Initialize with the first element
    scalar_t max_val = input[base];

    // Use loop unrolling with a factor of 4. The loop starts at index 1 because index 0 is already used.
    int unroll_end = 1 + ((dim_size - 1) / 4) * 4;
    
    #pragma unroll
    for (int i = 1; i < unroll_end; i += 4) {
        scalar_t a = input[base + i * inner_size];
        scalar_t b = input[base + (i + 1) * inner_size];
        scalar_t c = input[base + (i + 2) * inner_size];
        scalar_t d = input[base + (i + 3) * inner_size];
        max_val = max(max_val, a);
        max_val = max(max_val, b);
        max_val = max(max_val, c);
        max_val = max(max_val, d);
    }

    // Process any remaining iterations
    #pragma unroll
    for (int i = unroll_end; i < dim_size; i++) {
        max_val = max(max_val, input[base + i * inner_size]);
    }

    output[idx] = max_val;
}

// CUDA forward function
torch::Tensor max_reduce_cuda_forward(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();

    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= input.size(i);
    }

    int64_t inner_size = 1;
    for (int i = dim + 1; i < input.dim(); i++) {
        inner_size *= input.size(i);
    }

    const int64_t dim_size = input.size(dim);
    
    // Compute output tensor sizes by removing the reduction dimension
    auto output_sizes = input.sizes().vec();
    output_sizes.erase(output_sizes.begin() + dim);
    auto output = torch::empty(output_sizes, input.options());

    const int threads = 256;
    const int blocks = (outer_size * inner_size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "max_reduce_unroll_forward", ([&] {
        max_reduce_unroll_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            dim_size,
            inner_size
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_reduce_cuda_forward, "Max reduce forward (CUDA) with loop unrolling");
}
