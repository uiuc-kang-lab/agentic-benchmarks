#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 32
#define BLOCK_PAD 1

template <typename scalar_t>
__global__ void matmul_cuda_kernel(const scalar_t* __restrict__ A, const scalar_t* __restrict__ B,
                                   scalar_t* __restrict__ C, int M, int K, int N) {
    // Padded shared memory to reduce bank conflicts
    __shared__ scalar_t sA[TILE_WIDTH][TILE_WIDTH + BLOCK_PAD];
    __shared__ scalar_t sB[TILE_WIDTH][TILE_WIDTH + BLOCK_PAD];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    
    // Register to accumulate results
    scalar_t sum = 0;

    // Loop over tiles
    for (int t = 0; t < (K + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {
        // Collaborative loading of tiles into shared memory
        if (row < M && t * TILE_WIDTH + threadIdx.x < K) {
            sA[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_WIDTH + threadIdx.x];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0;
        }

        if (t * TILE_WIDTH + threadIdx.y < K && col < N) {
            sB[threadIdx.y][threadIdx.x] = B[(t * TILE_WIDTH + threadIdx.y) * N + col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0;
        }

        __syncthreads();

        // Compute partial dot product
        #pragma unroll
        for (int i = 0; i < TILE_WIDTH; ++i) {
            sum += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Write result to global memory
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

torch::Tensor module_fn(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input tensor B must be a CUDA tensor");

    int64_t M = A.size(0);
    int64_t K = A.size(1);
    int64_t N = B.size(1);

    TORCH_CHECK(K == B.size(0), "Inner dimensions of A and B must match");

    auto C = torch::empty({M, N}, A.options());

    dim3 threads(TILE_WIDTH, TILE_WIDTH);
    dim3 blocks((N + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matmul_cuda_kernel", ([&] {
        matmul_cuda_kernel<scalar_t><<<blocks, threads>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, K, N);
    }));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &module_fn, "Matrix multiplication forward (CUDA)");
}