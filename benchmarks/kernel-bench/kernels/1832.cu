#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 128
#define NUM_STREAMS 4
#define UNROLL_FACTOR 4

__global__ void triangular_mm_kernel_unrolled(const float* __restrict__ A,
                                              const float* __restrict__ B,
                                              float* __restrict__ C,
                                              int N,
                                              int tile_row) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    row += tile_row;
    
    if (row < N && col < N) {
        float sum = 0.f;
        
        // Handle main loop with unrolling
        int k_start = col;
        int k_end = (row / UNROLL_FACTOR) * UNROLL_FACTOR;
        
        #pragma unroll
        for (int k = k_start; k < k_end; k += UNROLL_FACTOR) {
            sum += __ldg(&A[row * N + k]) * __ldg(&B[k * N + col]);
            sum += __ldg(&A[row * N + (k+1)]) * __ldg(&B[(k+1) * N + col]);
            sum += __ldg(&A[row * N + (k+2)]) * __ldg(&B[(k+2) * N + col]);
            sum += __ldg(&A[row * N + (k+3)]) * __ldg(&B[(k+3) * N + col]);
        }
        
        // Handle remaining elements
        for (int k = k_end; k <= row; ++k) {
            sum += __ldg(&A[row * N + k]) * __ldg(&B[k * N + col]);
        }
        
        C[row * N + col] = (row >= col) ? sum : 0.f;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);

    for (int tile_row = 0; tile_row < N; tile_row += TILE_SIZE) {
        int current_tile_size = min(TILE_SIZE, N - tile_row);
        dim3 numBlocks((N + threads - 1) / threads,
                      (current_tile_size + threads - 1) / threads);

        int stream_idx = (tile_row / TILE_SIZE) % NUM_STREAMS;
        
        triangular_mm_kernel_unrolled<<<numBlocks, threadsPerBlock, 0, streams[stream_idx]>>>(
            A.data_ptr<float>(),
            B.data_ptr<float>(),
            C.data_ptr<float>(),
            N,
            tile_row
        );
    }

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}