#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   const int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col <= row) {
        float sum = 0.0f;
        for (int k = col; k <= row; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }

    if (row < N && col > row) {
        C[row * N + col] = 0.0f;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    const int threadsPerBlock = 16;
    dim3 threads(threadsPerBlock, threadsPerBlock);
    dim3 blocks((N + threadsPerBlock - 1) / threadsPerBlock, (N + threadsPerBlock - 1) / threadsPerBlock);

    triangular_mm_kernel<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}