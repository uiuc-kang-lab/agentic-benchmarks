#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Grid-stride loop based 2D convolution kernel
__global__ void grid_stride_conv2d_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int input_height,
    int input_width,
    int out_channels,
    int kernel_h,
    int kernel_w,
    int height_out,
    int width_out,
    int stride,
    int pad_h,
    int pad_w,
    int dilation_h,
    int dilation_w) 
{
    // Total number of output elements
    int total_elements = batch_size * out_channels * height_out * width_out;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_stride = blockDim.x * gridDim.x;

    // Each thread handles multiple output elements using a grid-stride loop
    for (int i = idx; i < total_elements; i += grid_stride) {
        int index = i;
        // Derive output coordinates: b, oc, h, w from the flattened index
        int w = index % width_out;
        index /= width_out;
        int h = index % height_out;
        index /= height_out;
        int oc = index % out_channels;
        int b = index / out_channels;

        float sum = (bias_ptr != nullptr) ? bias_ptr[oc] : 0.0f;

        // Compute the origin position in input with padding and stride
        int h_in_origin = h * stride - pad_h;
        int w_in_origin = w * stride - pad_w;

        // Loop over input channels and kernel window
        for (int ic = 0; ic < in_channels; ++ic) {
            for (int kh = 0; kh < kernel_h; ++kh) {
                int h_in = h_in_origin + kh * dilation_h;
                if (h_in < 0 || h_in >= input_height)
                    continue;
                for (int kw = 0; kw < kernel_w; ++kw) {
                    int w_in = w_in_origin + kw * dilation_w;
                    if (w_in < 0 || w_in >= input_width)
                        continue;

                    int x_index = b * in_channels * input_height * input_width +
                                  ic * input_height * input_width +
                                  h_in * input_width + w_in;
                    int w_index = oc * in_channels * kernel_h * kernel_w +
                                  ic * kernel_h * kernel_w +
                                  kh * kernel_w + kw;
                    sum += x[x_index] * weight[w_index];
                }
            }
        }

        // Write the computed value to the output tensor
        output[i] = sum;
    }
}

// Forward function exposed to PyTorch
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    std::tuple<int, int> padding,
    std::tuple<int, int> dilation) {

    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");

    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        TORCH_CHECK(bias->is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        bias_ptr = bias->data_ptr<float>();
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int input_height = x.size(2);
    int input_width = x.size(3);

    int out_channels = weight.size(0);
    int kernel_h = weight.size(2);
    int kernel_w = weight.size(3);

    int pad_h = std::get<0>(padding);
    int pad_w = std::get<1>(padding);
    int dilation_h = std::get<0>(dilation);
    int dilation_w = std::get<1>(dilation);

    int height_out = (input_height + 2 * pad_h - dilation_h * (kernel_h - 1) - 1) / stride + 1;
    int width_out = (input_width + 2 * pad_w - dilation_w * (kernel_w - 1) - 1) / stride + 1;

    auto output = torch::empty({batch_size, out_channels, height_out, width_out}, x.options());

    int total_elements = batch_size * out_channels * height_out * width_out;
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;

    grid_stride_conv2d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        input_height,
        input_width,
        out_channels,
        kernel_h,
        kernel_w,
        height_out,
        width_out,
        stride,
        pad_h,
        pad_w,
        dilation_h,
        dilation_w
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Conv2D forward (CUDA)");
}
