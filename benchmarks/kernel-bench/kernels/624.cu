#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH_32 32

// CUDA kernel for matrix multiplication using shared memory tiling
template <typename scalar_t>
__global__ void matmul_cuda_kernel_tuned(const scalar_t* __restrict__ A, const scalar_t* __restrict__ B,
                                   scalar_t* __restrict__ C, int M, int K, int N) {
    __shared__ scalar_t sA[TILE_WIDTH_32][TILE_WIDTH_32];
    __shared__ scalar_t sB[TILE_WIDTH_32][TILE_WIDTH_32];

    int row = blockIdx.y * TILE_WIDTH_32 + threadIdx.y; // M dimension
    int col = blockIdx.x * TILE_WIDTH_32 + threadIdx.x; // N dimension

    scalar_t value = 0;

    // Loop over tiles
    for (int t = 0; t < (K + TILE_WIDTH_32 - 1) / TILE_WIDTH_32; ++t) {
        // Load elements into shared memory
        if (row < M && t * TILE_WIDTH_32 + threadIdx.x < K)
            sA[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_WIDTH_32 + threadIdx.x];
        else
            sA[threadIdx.y][threadIdx.x] = 0;

        if (col < N && t * TILE_WIDTH_32 + threadIdx.y < K)
            sB[threadIdx.y][threadIdx.x] = B[(t * TILE_WIDTH_32 + threadIdx.y) * N + col];
        else
            sB[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();

        for (int i = 0; i < TILE_WIDTH_32; ++i) {
            value += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Write to output
    if (row < M && col < N) {
        C[row * N + col] = value;
    }
}

// Forward function
torch::Tensor module_fn(torch::Tensor A, torch::Tensor B) {
    // Ensure input tensors are CUDA tensors
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input tensor B must be a CUDA tensor");

    // Get matrix dimensions
    int64_t M = A.size(0);
    int64_t K = A.size(1);
    int64_t N = B.size(1);

    // Check dimensions compatibility
    TORCH_CHECK(K == B.size(0), "Inner dimensions of A and B must match");

    // Allocate output tensor
    auto C = torch::empty({M, N}, A.options());

    // Define block and grid dimensions
    dim3 threads_per_block(TILE_WIDTH_32, TILE_WIDTH_32);
    dim3 num_blocks((N + TILE_WIDTH_32 - 1) / TILE_WIDTH_32, (M + TILE_WIDTH_32 - 1) / TILE_WIDTH_32);

    // Launch the CUDA kernel
    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matmul_cuda_kernel_tuned", ([&] {
        matmul_cuda_kernel_tuned<scalar_t><<<num_blocks, threads_per_block>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, K, N);
    }));

    // Wait for all kernels to finish
    hipDeviceSynchronize();

    return C;
}

// Binding code
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &module_fn, "Matrix multiplication forward (CUDA)");
}