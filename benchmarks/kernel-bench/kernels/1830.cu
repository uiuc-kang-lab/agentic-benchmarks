#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 128
#define NUM_STREAMS 4

__global__ void triangular_mm_kernel_stride(const float* __restrict__ A,
                                            const float* __restrict__ B,
                                            float* __restrict__ C,
                                            int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loops to handle large workloads
    for (int stride_row = row; stride_row < N; stride_row += blockDim.y * gridDim.y) {
        for (int stride_col = col; stride_col <= stride_row; stride_col += blockDim.x * gridDim.x) {
            if (stride_row < N && stride_col < N) {
                float sum = 0.f;
                for (int k = stride_col; k <= stride_row; ++k) {
                    sum += __ldg(&A[stride_row * N + k]) * __ldg(&B[k * N + stride_col]);
                }
                C[stride_row * N + stride_col] = sum;
            }
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Create CUDA streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threads - 1) / threads, (N + threads - 1) / threads);

    // Launch kernel with stride loops
    triangular_mm_kernel_stride<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    // Synchronize all streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}
