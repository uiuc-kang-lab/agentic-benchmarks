#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 16

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   const int N) {
    // Calculate global thread indices
    const int global_x = blockIdx.x * BLOCK_DIM_X + threadIdx.x;
    const int global_y = blockIdx.y * BLOCK_DIM_Y + threadIdx.y;
    
    // Each thread computes one element of C
    if (global_y < N && global_x < N) {
        const int row = global_y;
        const int col = global_x;
        
        // Only compute for lower triangular part
        if (row >= col) {
            float sum = 0.0f;
            
            // Calculate start and end points for dot product
            const int start_k = col;
            const int end_k = row;
            
            // Unrolled loop for better instruction-level parallelism
            int k = start_k;
            #pragma unroll 4
            for (; k <= end_k - 4; k += 4) {
                sum += A[row * N + k] * B[k * N + col];
                sum += A[row * N + (k+1)] * B[(k+1) * N + col];
                sum += A[row * N + (k+2)] * B[(k+2) * N + col];
                sum += A[row * N + (k+3)] * B[(k+3) * N + col];
            }
            
            // Handle remaining elements
            for (; k <= end_k; k++) {
                sum += A[row * N + k] * B[k * N + col];
            }
            
            C[row * N + col] = sum;
        } else {
            C[row * N + col] = 0.0f;
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    const int N = A.size(0);
    auto C = torch::empty_like(A);

    // Calculate grid dimensions
    dim3 threadsPerBlock(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 numBlocks(
        (N + BLOCK_DIM_X - 1) / BLOCK_DIM_X,
        (N + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y
    );

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}