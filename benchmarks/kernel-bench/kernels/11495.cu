#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void __launch_bounds__(256) kl_div_kernel_even_workload(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    extern __shared__ float partial_sums[];
    
    // Register-based accumulation
    float sum = 0.0f;
    
    // Vectorized memory access with loop unrolling for better instruction-level parallelism
    #pragma unroll
    for (int i = 0; i < 4; ++i, idx += blockDim.x) {
        if (idx < n) {
            float log_pred = __ldg(&log_predictions[idx]);  // Use read-only cache
            float target = __ldg(&targets[idx]);
            sum += expf(log_pred) - target * log_pred;
        }
    }

    // Store sum in shared memory
    partial_sums[threadIdx.x] = sum;
    __syncthreads();

    // Uniform block reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            partial_sums[threadIdx.x] += partial_sums[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        atomicAdd(output, partial_sums[0]);
}

torch::Tensor kl_div_cuda_forward_even_workload(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    const int threads = 256;
    const int blocks = (n + threads*4 - 1) / (threads*4);
    const int shared_mem = threads * sizeof(float);
    
    kl_div_kernel_even_workload<<<blocks, threads, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward_even_workload, "KL divergence forward even workload (CUDA)");
}