#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32  // Increased tile size for better occupancy
#define WARP_SIZE 32

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   const int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    // Early exit if above diagonal
    if (row < col || row >= N || col >= N) {
        return;
    }

    float sum = 0.0f;
    
    // Calculate the starting tile for this thread
    const int start_tile = col / TILE_SIZE;
    const int end_tile = row / TILE_SIZE;
    
    #pragma unroll 1
    for (int t = start_tile; t <= end_tile; t++) {
        // Collaborative loading with vectorized memory access
        if (row < N && (t * TILE_SIZE + tx) <= row) {
            As[ty][tx] = __ldg(&A[row * N + (t * TILE_SIZE + tx)]);
        } else {
            As[ty][tx] = 0.0f;
        }
        
        if ((t * TILE_SIZE + ty) < N && col < N) {
            Bs[ty][tx] = __ldg(&B[(t * TILE_SIZE + ty) * N + col]);
        } else {
            Bs[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute partial sum for this tile using warp-level optimizations
        if ((t * TILE_SIZE) <= row) {
            const int k_start = (t == start_tile) ? col - (t * TILE_SIZE) : 0;
            const int k_end = min(TILE_SIZE, row - (t * TILE_SIZE) + 1);
            
            #pragma unroll 8
            for (int k = k_start; k < k_end; k++) {
                sum = __fmaf_rn(As[ty][k], Bs[k][tx], sum);
            }
        }
        
        __syncthreads();
    }
    
    C[row * N + col] = sum;
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D tensors");
    TORCH_CHECK(A.size(0) == A.size(1) && B.size(0) == B.size(1), "A and B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    const int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE, 
                   (N + TILE_SIZE - 1) / TILE_SIZE);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    triangular_mm_kernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized triangular matrix multiplication (CUDA)");
}