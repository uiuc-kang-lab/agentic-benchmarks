#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel for lower triangular matrix multiplication with manual loop unrolling
__global__ void unrolled_triangular_mm_kernel(const float* __restrict__ A,
                                               const float* __restrict__ B,
                                               float* __restrict__ C,
                                               int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        // Only lower triangular part is computed
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            float sum = 0.f;
            int k = col;
            // Unroll loop in steps of 4 to reduce loop overhead.
            int limit = row - 3;  // Process groups of 4 if possible
            #pragma unroll
            for (; k <= limit; k += 4) {
                sum += A[row * N + k]     * B[k     * N + col];
                sum += A[row * N + k + 1] * B[(k+1) * N + col];
                sum += A[row * N + k + 2] * B[(k+2) * N + col];
                sum += A[row * N + k + 3] * B[(k+3) * N + col];
            }
            // Tail loop for remaining iterations
            for (; k <= row; ++k) {
                sum += A[row * N + k] * B[k * N + col];
            }
            C[row * N + col] = sum;
        }
    }
}

// C++ interface exposed to PyTorch
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Use a 16x16 block configuration
    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threads - 1) / threads, (N + threads - 1) / threads);

    unrolled_triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Unrolled triangular matrix multiplication (CUDA) kernel");
}
