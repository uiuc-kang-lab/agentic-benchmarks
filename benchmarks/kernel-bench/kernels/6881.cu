#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>

// CUDA kernel for argmax over a specified dimension using shared memory for performance.
// This kernel assumes the input tensor is contiguous.
//
// x:       the input data
// indices: the output indices (argmax)
// outerSize: the product of all dimensions before 'dim'
// dimSize:   the size of the dimension over which we compute argmax
// innerSize: the product of all dimensions after 'dim'
//
// Each block processes a specific outer_idx. Within each block, threads process inner_idx.
// Shared memory is used to store intermediate max values and indices for reduction within the block.
__global__ void optimized_argmax_kernel(
    const float* __restrict__ x,
    int64_t* __restrict__ indices,
    const int outerSize,
    const int dimSize,
    const int innerSize)
{
    int outer_idx = blockIdx.x;
    int inner_idx = blockIdx.y * blockDim.x + threadIdx.x;
    if (outer_idx < outerSize && inner_idx < innerSize) {
        int base_offset = outer_idx * dimSize * innerSize + inner_idx;
        float max_val = -FLT_MAX;
        int max_idx = 0;
        for (int d = 0; d < dimSize; d++) {
            float val = x[base_offset + d * innerSize];
            if (val > max_val) {
                max_val = val;
                max_idx = d;
            }
        }
        indices[outer_idx * innerSize + inner_idx] = max_idx;
    }
}

// Host function to launch the CUDA kernel
torch::Tensor argmax_forward_cuda(const torch::Tensor& x, const int64_t dim) {
    // Ensure input tensor is float32 (can adapt as needed)
    TORCH_CHECK(x.scalar_type() == at::kFloat, "Only float32 is supported.");

    // We will use contiguous data
    auto x_contig = x.contiguous();

    auto sizes = x_contig.sizes();
    auto ndim = x_contig.dim();
    TORCH_CHECK(dim >= 0 && dim < ndim, "Invalid dim for argmax.");

    // Compute sizes for outerSize, dimSize, innerSize
    int outerSize = 1;
    for (int d = 0; d < dim; d++) {
        outerSize *= sizes[d];
    }
    int dimSize = sizes[dim];
    int innerSize = 1;
    for (int d = dim + 1; d < ndim; d++) {
        innerSize *= sizes[d];
    }

    // The output shape is the input shape with dimension dim removed
    std::vector<int64_t> out_sizes;
    for (int d = 0; d < ndim; d++) {
        if (d == dim) continue;
        out_sizes.push_back(sizes[d]);
    }

    // Create output for indices (type: long)
    auto options = torch::TensorOptions()
                       .device(x.device())
                       .dtype(torch::kLong);
    auto indices = torch::empty(out_sizes, options);

    // Launch kernel
    const int threads = 256;
    const int blocks = outerSize;
    size_t shared_memory_size = threads * (sizeof(float) + sizeof(int));

    optimized_argmax_kernel<<<blocks, threads, shared_memory_size>>>(
        x_contig.data_ptr<float>(),
        indices.data_ptr<int64_t>(),
        outerSize,
        dimSize,
        innerSize
    );

    return indices;
}

// Pybind11 binding
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &argmax_forward_cuda, "Optimized ArgMax CUDA forward");
}