#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cudnn/Handles.h>
#include <ATen/cudnn/Descriptors.h>
#include <hipDNN.h>

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
#define BLOCK_SIZE_Z 4

// Helper function to map at::ScalarType to hipdnnDataType_t
hipdnnDataType_t getCudnnDataType(at::ScalarType type) {
    switch (type) {
        case at::ScalarType::Float:
            return HIPDNN_DATA_FLOAT;
        case at::ScalarType::Double:
            return HIPDNN_DATA_DOUBLE;
        case at::ScalarType::Half:
            return HIPDNN_DATA_HALF;
        default:
            TORCH_CHECK(false, "Unsupported data type for cuDNN");
    }
}

__global__ void conv3d_shared_memory_kernel(
    float* __restrict__ output,
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    int batch_size, int in_channels, int out_channels,
    int in_depth, int in_height, int in_width,
    int kernel_d, int kernel_h, int kernel_w,
    int out_depth, int out_height, int out_width,
    int stride, int padding, int dilation, int groups) {

    extern __shared__ float shared_mem[];
    float* shared_input = shared_mem;
    float* shared_weight = shared_mem + BLOCK_SIZE_X * BLOCK_SIZE_Y * BLOCK_SIZE_Z;

    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;
    int tid_z = threadIdx.z;

    int w_out = blockIdx.x * blockDim.x + tid_x;
    int h_out = blockIdx.y * blockDim.y + tid_y;
    int d_out = blockIdx.z * blockDim.z + tid_z;

    if (w_out >= out_width || h_out >= out_height || d_out >= out_depth) return;

    int c_out = blockIdx.z;
    int b = blockIdx.y;

    float sum = 0.0f;

    int group = c_out / (out_channels / groups);
    int in_channels_per_group = in_channels / groups;

    for (int ic = 0; ic < in_channels_per_group; ic++) {
        int in_c = group * in_channels_per_group + ic;

        for (int kd = 0; kd < kernel_d; kd++) {
            int d_in = d_out * stride - padding + kd * dilation;
            if (d_in < 0 || d_in >= in_depth) continue;

            for (int kh = 0; kh < kernel_h; kh++) {
                int h_in = h_out * stride - padding + kh * dilation;
                if (h_in < 0 || h_in >= in_height) continue;

                for (int kw = 0; kw < kernel_w; kw++) {
                    int w_in = w_out * stride - padding + kw * dilation;
                    if (w_in < 0 || w_in >= in_width) continue;

                    int input_index = ((b * in_channels + in_c) * in_depth + d_in) * in_height * in_width +
                                      h_in * in_width + w_in;
                    int weight_index = (((c_out * in_channels_per_group) + ic) * kernel_d + kd) * kernel_h * kernel_w +
                                       kh * kernel_w + kw;

                    shared_input[tid_z * BLOCK_SIZE_Y * BLOCK_SIZE_X + tid_y * BLOCK_SIZE_X + tid_x] = input[input_index];
                    shared_weight[tid_z * BLOCK_SIZE_Y * BLOCK_SIZE_X + tid_y * BLOCK_SIZE_X + tid_x] = weight[weight_index];

                    __syncthreads();

                    sum += shared_input[tid_z * BLOCK_SIZE_Y * BLOCK_SIZE_X + tid_y * BLOCK_SIZE_X + tid_x] *
                           shared_weight[tid_z * BLOCK_SIZE_Y * BLOCK_SIZE_X + tid_y * BLOCK_SIZE_X + tid_x];

                    __syncthreads();
                }
            }
        }
    }

    if (bias != nullptr) {
        sum += bias[c_out];
    }

    int output_index = ((b * out_channels + c_out) * out_depth + d_out) * out_height * out_width +
                       h_out * out_width + w_out;
    output[output_index] = sum;
}

at::Tensor forward(
    const at::Tensor& input,
    const at::Tensor& weight,
    const c10::optional<at::Tensor>& bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups
) {
    auto bias = bias_opt.value_or(at::Tensor());
    // Ensure inputs are on CUDA
    TORCH_CHECK(input.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "Weight must be a CUDA tensor");
    TORCH_CHECK(!bias.defined() || bias.is_cuda(), "Bias must be a CUDA tensor");

    // Get input dimensions
    int64_t batch_size = input.size(0);
    int64_t in_channels = input.size(1);
    int64_t in_depth = input.size(2);
    int64_t in_height = input.size(3);
    int64_t in_width = input.size(4);

    // Get weight dimensions
    int64_t out_channels = weight.size(0);
    int64_t kernel_d = weight.size(2);
    int64_t kernel_h = weight.size(3);
    int64_t kernel_w = weight.size(4);

    // Calculate output dimensions
    int64_t out_depth = (in_depth + 2 * padding - dilation * (kernel_d - 1) - 1) / stride + 1;
    int64_t out_height = (in_height + 2 * padding - dilation * (kernel_h - 1) - 1) / stride + 1;
    int64_t out_width = (in_width + 2 * padding - dilation * (kernel_w - 1) - 1) / stride + 1;

    // Prepare output tensor
    auto options = input.options();
    auto output = at::empty({batch_size, out_channels, out_depth, out_height, out_width}, options);

    // Launch kernel
    dim3 threadsPerBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    dim3 numBlocks(
        (out_width + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X,
        (out_height + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y,
        (out_depth + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z
    );

    size_t shared_memory_size = (BLOCK_SIZE_X * BLOCK_SIZE_Y * BLOCK_SIZE_Z) * sizeof(float) * 2;

    conv3d_shared_memory_kernel<<<numBlocks, threadsPerBlock, shared_memory_size>>>(
        output.data_ptr<float>(),
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.defined() ? bias.data_ptr<float>() : nullptr,
        batch_size, in_channels, out_channels,
        in_depth, in_height, in_width,
        kernel_d, kernel_h, kernel_w,
        out_depth, out_height, out_width,
        stride, padding, dilation, groups
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D convolution forward using shared memory optimization (CUDA)");
}