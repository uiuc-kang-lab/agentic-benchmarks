#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>
#include <cmath>

// Combined kernel utilizing warp-level primitives for max reduction and unrolled reduction for sum of exponentials

template <typename scalar_t, int BLOCK_SIZE>
__global__ void efficient_logsoftmax_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int dim_size) {

    int batch_idx = blockIdx.x;
    const scalar_t* input_row = input + batch_idx * dim_size;
    scalar_t* output_row = output + batch_idx * dim_size;

    __shared__ scalar_t sdata[BLOCK_SIZE];

    // Phase 1: Compute the maximum value in the row using warp-level reduction
    scalar_t local_max = -std::numeric_limits<scalar_t>::infinity();
    for (int idx = threadIdx.x; idx < dim_size; idx += BLOCK_SIZE) {
        scalar_t val = input_row[idx];
        local_max = max(local_max, val);
    }

    unsigned int mask = 0xffffffff;
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        scalar_t other = __shfl_down_sync(mask, local_max, offset);
        local_max = max(local_max, other);
    }

    if (threadIdx.x % warpSize == 0) {
        sdata[threadIdx.x / warpSize] = local_max;
    }
    __syncthreads();

    if (threadIdx.x < BLOCK_SIZE / warpSize) {
        scalar_t block_max = -std::numeric_limits<scalar_t>::infinity();
        if (threadIdx.x < BLOCK_SIZE / warpSize) {
            block_max = max(block_max, sdata[threadIdx.x]);
        }
        sdata[threadIdx.x] = block_max;
    }
    __syncthreads();

    scalar_t max_val = sdata[0];

    // Phase 2: Compute the sum of exp(x - max_val) using warp-level reduction and unrolling
    scalar_t local_sum = 0;
    #pragma unroll
    for (int idx = threadIdx.x; idx < dim_size; idx += BLOCK_SIZE) {
        scalar_t exp_val = exp(input_row[idx] - max_val);
        local_sum += exp_val;
    }

    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        local_sum += __shfl_down_sync(mask, local_sum, offset);
    }

    if (threadIdx.x % warpSize == 0) {
        sdata[threadIdx.x / warpSize] = local_sum;
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        scalar_t block_sum = 0;
        for (int i = 0; i < BLOCK_SIZE / warpSize; ++i) {
            block_sum += sdata[i];
        }
        sdata[0] = block_sum;
    }
    __syncthreads();

    scalar_t sum = sdata[0];
    scalar_t log_sum = log(sum);

    // Phase 3: Write back the final LogSoftmax values
    for (int idx = threadIdx.x; idx < dim_size; idx += BLOCK_SIZE) {
        output_row[idx] = (input_row[idx] - max_val) - log_sum;
    }
}

// Host function

torch::Tensor efficient_logsoftmax_cuda_forward(torch::Tensor input, int64_t dim) {
    TORCH_CHECK(input.is_cuda(), "input must be a CUDA tensor");
    TORCH_CHECK(
        input.scalar_type() == torch::kFloat32 || input.scalar_type() == torch::kFloat64,
        "input must be float32 or float64");

    int64_t ndim = input.dim();
    TORCH_CHECK(dim >= -ndim && dim < ndim, "dim out of range");
    dim = dim >= 0 ? dim : dim + ndim;

    std::vector<int64_t> permute_dims;
    for (int64_t i = 0; i < ndim; ++i) {
        if (i != dim) {
            permute_dims.push_back(i);
        }
    }
    permute_dims.push_back(dim);

    input = input.permute(permute_dims).contiguous();
    int64_t batch_size = input.numel() / input.size(-1);
    int64_t dim_size = input.size(-1);

    auto output = torch::empty_like(input);

    int optimal_block_size = 256;
    if (dim_size <= 32) {
        optimal_block_size = 32;
    } else if (dim_size <= 64) {
        optimal_block_size = 64;
    } else if (dim_size <= 128) {
        optimal_block_size = 128;
    } else if (dim_size <= 256) {
        optimal_block_size = 256;
    } else if (dim_size <= 512) {
        optimal_block_size = 512;
    } else {
        optimal_block_size = 512;
    }

    const int blocks = batch_size;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "efficient_logsoftmax_cuda_forward", ([&] {
        if (optimal_block_size == 32) {
            efficient_logsoftmax_kernel<scalar_t, 32><<<blocks, 32>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else if (optimal_block_size == 64) {
            efficient_logsoftmax_kernel<scalar_t, 64><<<blocks, 64>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else if (optimal_block_size == 128) {
            efficient_logsoftmax_kernel<scalar_t, 128><<<blocks, 128>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else if (optimal_block_size == 256) {
            efficient_logsoftmax_kernel<scalar_t, 256><<<blocks, 256>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else if (optimal_block_size == 512) {
            efficient_logsoftmax_kernel<scalar_t, 512><<<blocks, 512>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        }
    }));

    std::vector<int64_t> inverse_permute_dims(ndim);
    for (size_t i = 0; i < permute_dims.size(); ++i) {
        inverse_permute_dims[permute_dims[i]] = i;
    }
    output = output.permute(inverse_permute_dims);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &efficient_logsoftmax_cuda_forward, "Efficient LogSoftmax forward (CUDA)");
}
