#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Experiment with block sizes; optimal configuration found: 128 threads per block
#define BLOCK_SIZE 128

__global__ void softmax_kernel(const float* __restrict__ x, float* __restrict__ y, int num_features) {
    int batch_idx = blockIdx.x;
    int tid = threadIdx.x;
    int stride = blockDim.x;

    const float* x_row = x + batch_idx * num_features;
    float* y_row = y + batch_idx * num_features;

    extern __shared__ float sdata[];

    // Pass 1: Compute maximum value of the row
    float thread_max = -INFINITY;
    for (int i = tid; i < num_features; i += stride) {
        thread_max = max(thread_max, x_row[i]);
    }
    sdata[tid] = thread_max;
    __syncthreads();

    // Reduction for maximum
    for (unsigned int s = stride / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = max(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }
    float max_val = sdata[0];

    // Pass 2: Compute exponentials and partial sum
    float thread_sum = 0.0f;
    for (int i = tid; i < num_features; i += stride) {
        float exp_val = __expf(x_row[i] - max_val);
        y_row[i] = exp_val;  // store temporary exponential
        thread_sum += exp_val;
    }
    sdata[tid] = thread_sum;
    __syncthreads();

    // Reduction for sum
    for (unsigned int s = stride / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    float sum_val = sdata[0];

    // Pass 3: Normalize the values
    for (int i = tid; i < num_features; i += stride) {
        y_row[i] = y_row[i] / sum_val;
    }
}

void softmax_forward_cuda(const float* x, float* y, int batch_size, int num_features) {
    dim3 grid_dim(batch_size);
    dim3 block_dim(BLOCK_SIZE);
    int shared_mem_size = BLOCK_SIZE * sizeof(float);
    
    softmax_kernel<<<grid_dim, block_dim, shared_mem_size>>>(x, y, num_features);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in softmax_forward_cuda: %s\n", hipGetErrorString(err));
        return;
    }
}

// C++ forward function
torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor.");
    TORCH_CHECK(x.dim() == 2, "Input tensor must be 2D.");
    TORCH_CHECK(x.scalar_type() == torch::kFloat32, "Input tensor must be float32.");

    int batch_size = x.size(0);
    int num_features = x.size(1);

    auto y = torch::empty_like(x);

    softmax_forward_cuda(x.data_ptr<float>(), y.data_ptr<float>(), batch_size, num_features);
    
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Softmax forward (CUDA)");
}
