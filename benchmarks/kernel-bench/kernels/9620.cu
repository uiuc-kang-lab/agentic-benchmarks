#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

// Optimized depthwise 2D convolution kernel using shared memory and warp-level primitives.
template <typename scalar_t>
__global__ void depthwiseConv2DSharedKernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ w,
    const scalar_t* __restrict__ b,
    scalar_t* __restrict__ out,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int kernel_size,
    const int out_height,
    const int out_width,
    const int stride,
    const int padding)
{
    extern __shared__ scalar_t shared_mem[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch_size * in_channels * out_height * out_width;
    if (idx >= total) {
        return;
    }

    // Decompose idx into (n, c, h_out, w_out).
    int w_out_idx = idx % out_width;
    int tmp = idx / out_width;
    int h_out_idx = tmp % out_height;
    tmp /= out_height;
    int c = tmp % in_channels;
    int n = tmp / in_channels;

    // Load input and weights into shared memory
    int thread_id = threadIdx.x;
    int num_threads = blockDim.x;
    for (int i = thread_id; i < kernel_size * kernel_size; i += num_threads) {
        shared_mem[i] = w[c * kernel_size * kernel_size + i];
    }
    __syncthreads();

    // Accumulate over the kernel.
    scalar_t value = 0;
    for (int kh = 0; kh < kernel_size; kh++) {
        for (int kw = 0; kw < kernel_size; kw++) {
            int h_in = h_out_idx * stride - padding + kh;
            int w_in = w_out_idx * stride - padding + kw;
            // Boundary check.
            if (h_in >= 0 && h_in < in_height && w_in >= 0 && w_in < in_width) {
                int x_index = ((n * in_channels + c) * in_height + h_in) * in_width + w_in;
                int w_index = kh * kernel_size + kw;
                value += x[x_index] * shared_mem[w_index];
            }
        }
    }

    // Warp-level reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        value += __shfl_down_sync(0xffffffff, value, offset);
    }

    // Add bias for this channel.
    if (threadIdx.x % warpSize == 0) {
        out[idx] = value + b[c];
    }
}

// The actual implementation of depthwise Conv2D in CUDA.
torch::Tensor forward_impl(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int groups)
{
    // For depthwise conv: groups == in_channels typically.
    // Compute output dimensions.
    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);

    const int kernel_size = weight.size(2);  // weight is (in_channels, 1, K, K)
    // Output height/width formula for convolution.
    const int out_height = (in_height + 2 * padding - kernel_size) / stride + 1;
    const int out_width  = (in_width  + 2 * padding - kernel_size) / stride + 1;

    // Create output tensor.
    auto out = torch::empty({batch_size, in_channels, out_height, out_width}, x.options());

    // Launch kernel.
    const int total = batch_size * in_channels * out_height * out_width;
    const int threads = 256;
    const int blocks = (total + threads - 1) / threads;
    const int shared_mem_size = kernel_size * kernel_size * sizeof(scalar_t);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "depthwise_conv2d_forward", ([&] {
        depthwiseConv2DSharedKernel<scalar_t><<<blocks, threads, shared_mem_size>>>(
            x.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            bias.data_ptr<scalar_t>(),
            out.data_ptr<scalar_t>(),
            batch_size, in_channels, in_height, in_width,
            kernel_size, out_height, out_width,
            stride, padding
        );
    }));

    return out;
}

namespace py = pybind11;

torch::Tensor forward_wrap(
    torch::Tensor x,
    torch::Tensor weight,
    py::object bias_obj,
    int stride,
    int padding,
    int groups)
{
    torch::Tensor bias;
    if (bias_obj.is_none()) {
        bias = torch::zeros({x.size(1)}, x.options());
    } else {
        bias = bias_obj.cast<torch::Tensor>();
    }
    return forward_impl(x, weight, bias, stride, padding, groups);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        &forward_wrap,
        "Depthwise conv2d forward (handles optional bias)",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = 1,
        py::arg("padding") = 0,
        py::arg("groups") = 1
    );
}