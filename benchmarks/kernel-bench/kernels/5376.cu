#include "hip/hip_runtime.h"
/*
 * A combined optimized CUDA kernel for 2D max pooling, merging coalesced memory access
 * and loop unrolling for small kernel sizes.
 */

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>


// Optimized kernel: uses 2D grid for spatial coalescing and loop unrolling for kernel_size 2 & 3
template <typename scalar_t>
__global__ void max_pool2d_optimized_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    // Determine output spatial position from thread indices
    int ow = threadIdx.x + blockIdx.x * blockDim.x;
    int oh = threadIdx.y + blockIdx.y * blockDim.y;

    // Each block in z dimension represents a (batch, channel) pair
    int bc = blockIdx.z;
    if (ow >= output_width || oh >= output_height) return;

    int b = bc / channels;
    int c = bc % channels;

    // Compute the flat output index (row-major ordering: batch, channel, height, width)
    int out_idx = ((b * channels + c) * output_height + oh) * output_width + ow;

    // Initialize max value to negative infinity
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    // Determine the starting position in the input tensor
    int h_start = oh * stride - padding;
    int w_start = ow * stride - padding;

    // Use loop unrolling for common small kernel sizes
    if (kernel_size == 2) {
        #pragma unroll
        for (int kh = 0; kh < 2; kh++) {
            int ih = h_start + kh * dilation;
            if (ih < 0 || ih >= input_height) continue;
            #pragma unroll
            for (int kw = 0; kw < 2; kw++) {
                int iw = w_start + kw * dilation;
                if (iw < 0 || iw >= input_width) continue;
                int input_idx = ((b * channels + c) * input_height + ih) * input_width + iw;
                max_val = max(max_val, __ldg(&input[input_idx]));
            }
        }
    } else if (kernel_size == 3) {
        #pragma unroll
        for (int kh = 0; kh < 3; kh++) {
            int ih = h_start + kh * dilation;
            if (ih < 0 || ih >= input_height) continue;
            #pragma unroll
            for (int kw = 0; kw < 3; kw++) {
                int iw = w_start + kw * dilation;
                if (iw < 0 || iw >= input_width) continue;
                int input_idx = ((b * channels + c) * input_height + ih) * input_width + iw;
                max_val = max(max_val, __ldg(&input[input_idx]));
            }
        }
    } else {
        // General case for arbitrary kernel_size
        for (int kh = 0; kh < kernel_size; kh++) {
            int ih = h_start + kh * dilation;
            if (ih < 0 || ih >= input_height) continue;
            for (int kw = 0; kw < kernel_size; kw++) {
                int iw = w_start + kw * dilation;
                if (iw < 0 || iw >= input_width) continue;
                int input_idx = ((b * channels + c) * input_height + ih) * input_width + iw;
                max_val = max(max_val, __ldg(&input[input_idx]));
            }
        }
    }

    output[out_idx] = max_val;
}


// Host function to launch the optimized max pooling kernel
// Uses a 2D block configuration for spatial dimensions and a 3D grid for (batch, channel, spatial dims)

torch::Tensor max_pool2d_optimized_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const int batch_size = input.size(0);
    const int channels = input.size(1);
    const int input_height = input.size(2);
    const int input_width = input.size(3);

    // Calculate output dimensions
    const int output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const int output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    // Configure block and grid to enforce coalesced memory access.
    // 32 threads in x (matching the warp size) and 8 in y, similar to kernel1.
    dim3 block(32, 8); // Block size aligned to warp size for better performance
    dim3 grid(
        (output_width + block.x - 1) / block.x,
        (output_height + block.y - 1) / block.y,
        batch_size * channels
    );

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "max_pool2d_optimized_cuda_forward", ([&] {
        max_pool2d_optimized_kernel<scalar_t><<<grid, block>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_optimized_cuda_forward, "Max Pool 2D optimized forward (CUDA)");
}
