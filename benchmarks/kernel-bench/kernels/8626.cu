#include "hip/hip_runtime.h"
/*
 * Optimized Transposed Conv3D CUDA Kernel
 * This implementation merges the native extension wrapper (for argument checking and dimension extraction)
 * with a custom grid-stride kernel similar to the manual implementation. It precomputes common offsets
 * (e.g., output base coordinates) to reduce redundant arithmetic and improve memory access efficiency.
 */

#include <torch/extension.h>
#include <vector>

// Macros to check tensor properties
#define CHECK_CUDA(x) TORCH_CHECK((x).is_cuda(), #x " must be a CUDA tensor");
#define CHECK_CONTIGUOUS(x) TORCH_CHECK((x).is_contiguous(), #x " must be contiguous");
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x);


// Optimized CUDA kernel using grid-stride loops and precomputation to reduce overhead
__global__ void transposed_conv3d_opt_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias, // may be nullptr
    float* __restrict__ output,
    int batch,
    int in_channels,
    int in_d,
    int in_h,
    int in_w,
    int out_channels,
    int out_d,
    int out_h,
    int out_w,
    int k_d,
    int k_h,
    int k_w,
    int s_d,
    int s_h,
    int s_w,
    int p_d,
    int p_h,
    int p_w,
    int groups,
    int channels_per_group_in,
    int channels_per_group_out) {

  const int total = batch * out_channels * out_d * out_h * out_w;
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < total;
       idx += blockDim.x * gridDim.x) {

    // Decode the flat index into 5D coordinates
    int tmp = idx;
    int w_out = tmp % out_w; tmp /= out_w;
    int h_out = tmp % out_h; tmp /= out_h;
    int d_out = tmp % out_d; tmp /= out_d;
    int oc = tmp % out_channels; tmp /= out_channels;
    int n = tmp;  

    // Initialize accumulation with bias if provided
    float sum = (bias != nullptr) ? bias[oc] : 0.0f;

    // Determine group and intra-group channel index
    int group = oc / channels_per_group_out;
    int oc_in_group = oc % channels_per_group_out;

    // Precompute base coordinates for the output location (incorporating padding)
    int d_base = d_out + p_d;
    int h_base = h_out + p_h;
    int w_base = w_out + p_w;

    // Loop over kernel dimensions
    for (int kd = 0; kd < k_d; kd++) {
      int tmp_d = d_base - kd;
      if (tmp_d % s_d != 0) continue;  // Only proceed if aligned with stride
      int in_d_idx = tmp_d / s_d;
      if (in_d_idx < 0 || in_d_idx >= in_d) continue;

      for (int kh = 0; kh < k_h; kh++) {
        int tmp_h = h_base - kh;
        if (tmp_h % s_h != 0) continue;
        int in_h_idx = tmp_h / s_h;
        if (in_h_idx < 0 || in_h_idx >= in_h) continue;

        for (int kw = 0; kw < k_w; kw++) {
          int tmp_w = w_base - kw;
          if (tmp_w % s_w != 0) continue;
          int in_w_idx = tmp_w / s_w;
          if (in_w_idx < 0 || in_w_idx >= in_w) continue;

          // For this kernel offset, iterate over the input channels in the current group
          for (int ic = 0; ic < channels_per_group_in; ic++) {
            int in_channel = group * channels_per_group_in + ic;
            // Compute flattened index for input: [n, in_channel, in_d_idx, in_h_idx, in_w_idx]
            int input_idx = n * (in_channels * in_d * in_h * in_w) +
                            in_channel * (in_d * in_h * in_w) +
                            in_d_idx * (in_h * in_w) +
                            in_h_idx * in_w + in_w_idx;
            float in_val = input[input_idx];

            // Compute flattened index for weight: [in_channel, oc_in_group, kd, kh, kw]
            int weight_idx = in_channel * (channels_per_group_out * k_d * k_h * k_w) +
                             oc_in_group * (k_d * k_h * k_w) +
                             kd * (k_h * k_w) + kh * k_w + kw;
            float wt = weight[weight_idx];

            sum += in_val * wt;
          } // end ic loop
        } // end kw loop
      } // end kh loop
    } // end kd loop

    // Write the result to the output tensor
    output[idx] = sum;
  }
}

// C++ forward function wrapping the custom CUDA kernel
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias_opt,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> output_padding,
    int64_t groups) {

  CHECK_INPUT(x);
  CHECK_INPUT(weight);
  if (bias_opt.has_value()) {
    CHECK_INPUT(*bias_opt);
  }

  // Extract input dimensions
  const int batch = x.size(0);
  const int in_channels = x.size(1);
  const int in_d = x.size(2);
  const int in_h = x.size(3);
  const int in_w = x.size(4);

  // Extract kernel dimensions from weight (assumed shape: [in_channels, out_channels_per_group, k_d, k_h, k_w])
  const int k_d = weight.size(2);
  const int k_h = weight.size(3);
  const int k_w = weight.size(4);

  // Strides and padding
  const int s_d = stride[0];
  const int s_h = stride[1];
  const int s_w = stride[2];
  const int p_d = padding[0];
  const int p_h = padding[1];
  const int p_w = padding[2];
  const int op_d = output_padding[0];
  const int op_h = output_padding[1];
  const int op_w = output_padding[2];

  // Compute output dimensions according to the transposed conv3d formula
  const int out_d = (in_d - 1) * s_d - 2 * p_d + k_d + op_d;
  const int out_h = (in_h - 1) * s_h - 2 * p_h + k_h + op_h;
  const int out_w = (in_w - 1) * s_w - 2 * p_w + k_w + op_w;

  // Determine output channels (weight shape: [in_channels, out_channels_per_group, k_d, k_h, k_w])
  const int channels_per_group_out = weight.size(1);
  const int out_channels = channels_per_group_out * groups;
  const int channels_per_group_in = in_channels / groups;

  // Allocate output tensor
  auto output = torch::zeros({batch, out_channels, out_d, out_h, out_w}, x.options());

  // Get raw pointers
  const float* x_ptr = x.data_ptr<float>();
  const float* weight_ptr = weight.data_ptr<float>();
  const float* bias_ptr = bias_opt.has_value() ? (*bias_opt).data_ptr<float>() : nullptr;
  float* out_ptr = output.data_ptr<float>();

  // Total number of output elements
  const int total = batch * out_channels * out_d * out_h * out_w;
  const int threads = 256;
  const int blocks = (total + threads - 1) / threads;

  // Launch the optimized CUDA kernel
  transposed_conv3d_opt_kernel<<<blocks, threads>>>(
      x_ptr,
      weight_ptr,
      bias_ptr,
      out_ptr,
      batch,
      in_channels,
      in_d,
      in_h,
      in_w,
      out_channels,
      out_d,
      out_h,
      out_w,
      k_d,
      k_h,
      k_w,
      s_d,
      s_h,
      s_w,
      p_d,
      p_h,
      p_w,
      groups,
      channels_per_group_in,
      channels_per_group_out);

  // Check for kernel launch errors
  hipError_t err = hipGetLastError();
  TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

  return output;
}

// PyBind11 module definition
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "Optimized Transposed Conv3D forward (CUDA)");
}
