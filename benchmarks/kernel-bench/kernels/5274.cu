#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void max_pool1d_flattened_kernel(
    const float* __restrict__ input,
    float* output,
    int64_t* indices,
    const int batch_size,
    const int num_channels,
    const int input_length,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation,
    const int output_length,
    bool return_indices)
{
    const int total_threads = batch_size * num_channels * output_length;
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid >= total_threads) return;
    
    const int b = tid / (num_channels * output_length);
    const int remaining = tid % (num_channels * output_length);
    const int c = remaining / output_length;
    const int i = remaining % output_length;
    
    const int input_start = i * stride - padding;
    float max_val = -INFINITY;
    int max_idx = -1;

    for (int k = 0; k < kernel_size; ++k) {
        const int pos = input_start + k * dilation;
        if (pos >= 0 && pos < input_length) {
            const float val = input[b * num_channels * input_length + c * input_length + pos];
            if (val > max_val) {
                max_val = val;
                max_idx = pos;
            }
        }
    }

    const int out_idx = b * num_channels * output_length + c * output_length + i;
    output[out_idx] = max_val;
    if (return_indices) indices[out_idx] = max_idx;
}

torch::Tensor forward(
    torch::Tensor x,
    int64_t kernel_size,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    bool return_indices)
{
    TORCH_CHECK(x.dim() == 3, "Input must be 3D");
    TORCH_CHECK(x.is_cuda(), "Input must be on CUDA");
    TORCH_CHECK(x.is_contiguous(), "Input must be contiguous");

    const int batch_size = x.size(0);
    const int num_channels = x.size(1);
    const int input_length = x.size(2);

    const int output_length = ((input_length + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    TORCH_CHECK(output_length > 0, "Output length must be positive");

    auto options = torch::TensorOptions().dtype(x.dtype()).device(x.device());
    auto output = torch::empty({batch_size, num_channels, output_length}, options);
    torch::Tensor indices;

    if (return_indices) {
        indices = torch::empty({batch_size, num_channels, output_length}, 
            options.dtype(torch::kInt64));
    }

    const int total_elements = batch_size * num_channels * output_length;
    const int threads_per_block = 256;
    const int num_blocks = (total_elements + threads_per_block - 1) / threads_per_block;

    max_pool1d_flattened_kernel<<<num_blocks, threads_per_block>>>(
        x.data_ptr<float>(),
        output.data_ptr<float>(),
        return_indices ? indices.data_ptr<int64_t>() : nullptr,
        batch_size,
        num_channels,
        input_length,
        kernel_size,
        stride,
        padding,
        dilation,
        output_length,
        return_indices
    );

    return return_indices ? torch::cat({output, indices}, -1) : output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "MaxPool1D forward (CUDA)");
}