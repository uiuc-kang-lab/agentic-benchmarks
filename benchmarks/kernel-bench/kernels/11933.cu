#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Warp-level reduction using shuffle instructions
template <typename scalar_t>
__device__ __forceinline__ scalar_t warp_reduce_sum(scalar_t val) {
    #pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

// Kernel with uniform control flow
template <typename scalar_t>
__global__ void triplet_margin_loss_kernel(
    const scalar_t* __restrict__ anchor,
    const scalar_t* __restrict__ positive,
    const scalar_t* __restrict__ negative,
    scalar_t* __restrict__ output,
    const float margin,
    const int batch_size,
    const int feat_size) {
    
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int warp_id = tid / 32;
    const int lane_id = tid % 32;
    const int sample_idx = warp_id;
    
    if (sample_idx >= batch_size) return;
    
    scalar_t dist_pos = 0;
    scalar_t dist_neg = 0;
    
    // Process features in a coalesced manner
    for (int i = lane_id; i < feat_size; i += 32) {
        const int idx = sample_idx * feat_size + i;
        const scalar_t a = anchor[idx];
        const scalar_t p = positive[idx];
        const scalar_t n = negative[idx];
        
        const scalar_t d_pos = a - p;
        const scalar_t d_neg = a - n;
        
        dist_pos += d_pos * d_pos;
        dist_neg += d_neg * d_neg;
    }
    
    // Warp-level reduction
    dist_pos = warp_reduce_sum(dist_pos);
    dist_neg = warp_reduce_sum(dist_neg);
    
    // Uniform control flow for final loss computation
    scalar_t loss = 0;
    if (lane_id == 0) {
        loss = sqrt(dist_pos) - sqrt(dist_neg) + margin;
    }
    loss = max(scalar_t(0.0), loss);
    if (lane_id == 0) {
        output[sample_idx] = loss;
    }
}

// CUDA entry point
torch::Tensor triplet_margin_loss_cuda(
    torch::Tensor anchor,
    torch::Tensor positive,
    torch::Tensor negative,
    float margin) {
    
    TORCH_CHECK(anchor.device().is_cuda(), "anchor must be a CUDA tensor");
    TORCH_CHECK(positive.device().is_cuda(), "positive must be a CUDA tensor");
    TORCH_CHECK(negative.device().is_cuda(), "negative must be a CUDA tensor");
    
    const int batch_size = anchor.size(0);
    const int feat_size = anchor.size(1);
    
    auto output = torch::zeros({batch_size}, anchor.options());
    
    const int threads_per_block = 128;
    const int warps_per_block = threads_per_block / 32;
    const int num_blocks = (batch_size + warps_per_block - 1) / warps_per_block;
    
    AT_DISPATCH_FLOATING_TYPES(anchor.scalar_type(), "triplet_margin_loss_kernel", ([&] {
        triplet_margin_loss_kernel<scalar_t><<<num_blocks, threads_per_block>>>(
            anchor.data_ptr<scalar_t>(),
            positive.data_ptr<scalar_t>(),
            negative.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            margin,
            batch_size,
            feat_size);
    }));
    
    return output.mean();
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &triplet_margin_loss_cuda, "Triplet margin loss forward (CUDA)");
}