#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized 2D average pooling kernel with improved thread and block indexing

template <typename scalar_t>
__global__ void avg_pool2d_forward_kernel_optimized(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int N,
    int C,
    int H,
    int W,
    int outH,
    int outW,
    int kernel_size,
    int stride,
    int padding
) {
    // Calculate the output index for this thread
    int w_out = blockIdx.x * blockDim.x + threadIdx.x;
    int h_out = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.z;
    int n = blockIdx.w;

    if (w_out >= outW || h_out >= outH) return;

    int h_start = h_out * stride - padding;
    int w_start = w_out * stride - padding;

    scalar_t sum_val = scalar_t(0);
    for (int i = 0; i < kernel_size; i++) {
        for (int j = 0; j < kernel_size; j++) {
            int h_in = h_start + i;
            int w_in = w_start + j;
            if (h_in >= 0 && h_in < H && w_in >= 0 && w_in < W) {
                sum_val += input[((n * C + c) * H + h_in) * W + w_in];
            }
        }
    }
    output[((n * C + c) * outH + h_out) * outW + w_out] = sum_val / static_cast<scalar_t>(kernel_size * kernel_size);
}

torch::Tensor avg_pool2d_forward_optimized(
    torch::Tensor x,
    int kernel_size,
    int stride,
    int padding
) {
    TORCH_CHECK(x.dim() == 4, "Input must be a 4D tensor.");
    auto N = x.size(0);
    auto C = x.size(1);
    auto H = x.size(2);
    auto W = x.size(3);

    int outH = (H + 2 * padding - kernel_size) / stride + 1;
    int outW = (W + 2 * padding - kernel_size) / stride + 1;

    auto x_cont = x.contiguous();
    auto options = x.options();
    auto out = torch::empty({N, C, outH, outW}, options);

    dim3 threads(16, 16);
    dim3 blocks((outW + threads.x - 1) / threads.x, (outH + threads.y - 1) / threads.y, C, N);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "avg_pool2d_forward_kernel_optimized", ([&] {
        const scalar_t* input_data = x_cont.data_ptr<scalar_t>();
        scalar_t* output_data = out.data_ptr<scalar_t>();

        avg_pool2d_forward_kernel_optimized<scalar_t><<<blocks, threads>>>(
            input_data,
            output_data,
            N, C, H, W,
            outH, outW,
            kernel_size, stride, padding
        );
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err));

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &avg_pool2d_forward_optimized, "2D Average Pooling forward with optimized indexing (CUDA)");
}