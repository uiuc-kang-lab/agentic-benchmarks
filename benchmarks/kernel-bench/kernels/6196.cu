#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device function to decompose a linear index into 5D coordinates
__device__ inline void decompose_index(int index, int out_w, int out_h, int out_d, int channels,
                                         int &n, int &c, int &d_out, int &h_out, int &w_out) {
    w_out = index % out_w;
    int tmp = index / out_w;
    h_out = tmp % out_h;
    tmp = tmp / out_h;
    d_out = tmp % out_d;
    tmp = tmp / out_d;
    c = tmp % channels;
    n = tmp / channels;
}

// Device function to compute the sum over a pooling window in the input
__device__ inline float compute_window_sum(const float* __restrict__ input,
                                              int n, int c,
                                              int d_out, int h_out, int w_out,
                                              int stride, int padding, int kernel_size,
                                              int in_d, int in_h, int in_w,
                                              int channels) {
    int d_start = d_out * stride - padding;
    int h_start = h_out * stride - padding;
    int w_start = w_out * stride - padding;

    int d_end = d_start + kernel_size;
    int h_end = h_start + kernel_size;
    int w_end = w_start + kernel_size;

    int d_start_clamped = (d_start < 0) ? 0 : d_start;
    int h_start_clamped = (h_start < 0) ? 0 : h_start;
    int w_start_clamped = (w_start < 0) ? 0 : w_start;
    int d_end_clamped = (d_end > in_d) ? in_d : d_end;
    int h_end_clamped = (h_end > in_h) ? in_h : h_end;
    int w_end_clamped = (w_end > in_w) ? in_w : w_end;

    float sum = 0.0f;
    for (int d = d_start_clamped; d < d_end_clamped; ++d) {
        for (int h = h_start_clamped; h < h_end_clamped; ++h) {
            for (int w = w_start_clamped; w < w_end_clamped; ++w) {
                int input_index = (((n * channels + c) * in_d + d) * in_h + h) * in_w + w;
                sum += input[input_index];
            }
        }
    }
    return sum;
}

// The modular 3D average pooling kernel
__global__ void avg_pool3d_modular_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int batch_size,
    int channels,
    int in_d, int in_h, int in_w,
    int out_d, int out_h, int out_w,
    int kernel_size, int stride, int padding) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = batch_size * channels * out_d * out_h * out_w;

    while (index < total_elements) {
        int n, c, d_out, h_out, w_out;
        decompose_index(index, out_w, out_h, out_d, channels, n, c, d_out, h_out, w_out);

        float sum = compute_window_sum(input, n, c, d_out, h_out, w_out,
                                         stride, padding, kernel_size,
                                         in_d, in_h, in_w, channels);
        int pool_volume = kernel_size * kernel_size * kernel_size;
        output[index] = sum / static_cast<float>(pool_volume);

        index += blockDim.x * gridDim.x;
    }
}

at::Tensor forward(at::Tensor input, int kernel_size, int stride, int padding) {
    // Ensure input is a 5D CUDA tensor
    TORCH_CHECK(input.dim() == 5, "Input tensor must be 5-dimensional");
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    int batch_size = input.size(0);
    int channels   = input.size(1);
    int in_d       = input.size(2);
    int in_h       = input.size(3);
    int in_w       = input.size(4);

    // Compute the output dimensions
    int out_d = (in_d + 2 * padding - kernel_size) / stride + 1;
    int out_h = (in_h + 2 * padding - kernel_size) / stride + 1;
    int out_w = (in_w + 2 * padding - kernel_size) / stride + 1;

    auto output = at::empty({batch_size, channels, out_d, out_h, out_w}, input.options());

    int total_elements = batch_size * channels * out_d * out_h * out_w;
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;

    const float* input_ptr = input.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();

    avg_pool3d_modular_kernel<<<blocks, threads>>>(input_ptr, output_ptr,
                                                    batch_size, channels,
                                                    in_d, in_h, in_w,
                                                    out_d, out_h, out_w,
                                                    kernel_size, stride, padding);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed : ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D Average Pooling forward (CUDA) - modular version");
}
