#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Define block dimensions for 2D tiling: threads per row and rows per block
#define THREADS_PER_ROW 256
#define ROWS_PER_BLOCK 4

// Kernel for matrix-vector multiplication using a 2D block configuration
// Each block processes ROWS_PER_BLOCK rows concurrently

template <typename scalar_t>
__global__ void matvec_mul_kernel_multirow(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ C,
    const int64_t M,
    const int64_t K) {

    // Compute the row index from 2D block indexing
    int row = blockIdx.x * ROWS_PER_BLOCK + threadIdx.y;
    if (row >= M) return;  // Out of bounds, exit

    // Each thread in the row computes a partial sum over columns
    scalar_t partial_sum = 0;
    for (int col = threadIdx.x; col < K; col += THREADS_PER_ROW) {
        // Use __ldg to utilize the read-only data cache
        partial_sum += __ldg(&A[row * K + col]) * __ldg(&B[col]);
    }

    // Use warp-level reduction to reduce partial sums within each warp.
    int lane = threadIdx.x & 31;  // lane index within the warp
    int warpId = threadIdx.x >> 5;  // warp index within the row
    constexpr int numWarps = THREADS_PER_ROW / 32;
    for (int offset = 16; offset > 0; offset >>= 1) {
        partial_sum += __shfl_down_sync(0xffffffff, partial_sum, offset);
    }

    // Allocate shared memory to store warp-level partial sums for each row.
    extern __shared__ scalar_t sdata[];
    if (lane == 0) {
        sdata[threadIdx.y * numWarps + warpId] = partial_sum;
    }
    __syncthreads();

    // The first thread of the first warp in each row accumulates the warp sums.
    if (threadIdx.x == 0) {
        scalar_t sum = 0;
        for (int i = 0; i < numWarps; i++) {
            sum += sdata[threadIdx.y * numWarps + i];
        }
        C[row] = sum;
    }
}

// C++ function wrapping the CUDA kernel
torch::Tensor matvec_mul_cuda(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");

    // Ensure contiguous layout for proper memory access
    A = A.contiguous();
    B = B.contiguous();

    const int64_t M = A.size(0);
    const int64_t K = A.size(1);

    // B should be a flat vector
    auto B_flat = B.view({-1});

    // Allocate output tensor for the result
    auto C = torch::zeros({M}, A.options());

    // Configure 2D block layout
    dim3 block(THREADS_PER_ROW, ROWS_PER_BLOCK);
    // Each block processes ROWS_PER_BLOCK rows, so grid size is computed accordingly
    int grid_x = (M + ROWS_PER_BLOCK - 1) / ROWS_PER_BLOCK;
    dim3 grid(grid_x);

    // Calculate shared memory size required per block
    size_t shared_mem_size = THREADS_PER_ROW * ROWS_PER_BLOCK * sizeof(float);
    // Launch kernel with AT_DISPATCH_FLOATING_TYPES to handle all floating point types
    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matvec_mul_cuda", ([&] {
        matvec_mul_kernel_multirow<scalar_t><<<grid, block, shared_mem_size>>>(
            A.data_ptr<scalar_t>(),
            B_flat.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M,
            K
        );
    }));

    return C.view({M, 1});
}

// PyBind11 binding code
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matvec_mul_cuda, "Matrix-Vector Multiplication (CUDA)");
}
