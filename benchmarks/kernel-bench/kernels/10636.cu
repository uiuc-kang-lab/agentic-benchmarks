#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Device function to compute cumulative product for a single sequence
// This function is modular and can be reused for different data types
// and configurations

template <typename scalar_t>
__device__ void compute_cumprod(
    scalar_t* output,
    const scalar_t* input,
    const int64_t base_offset,
    const int64_t dim_size,
    const int64_t stride) {
    scalar_t product = 1;
    for (int i = 0; i < dim_size; i++) {
        const int64_t offset = base_offset + i * stride;
        product *= input[offset];
        output[offset] = product;
    }
}

// Kernel function that utilizes the device function for cumulative product

template <typename scalar_t>
__global__ void cumprod_kernel_modular(
    scalar_t* output,
    const scalar_t* input,
    const int64_t numel,
    const int64_t dim_size,
    const int64_t stride) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int batch_idx = idx / stride;
    const int in_idx = idx % stride;
    
    if (idx < numel / dim_size) {
        const int64_t base_offset = batch_idx * (stride * dim_size) + in_idx;
        compute_cumprod(output, input, base_offset, dim_size, stride);
    }
}

torch::Tensor cumprod_cuda_forward_modular(torch::Tensor input, int64_t dim) {
    auto output = torch::empty_like(input);
    
    // Get tensor properties
    auto sizes = input.sizes();
    auto strides = input.strides();
    
    // Calculate dimension properties
    int64_t dim_size = sizes[dim];
    int64_t stride = strides[dim];
    int64_t numel = input.numel();
    
    // Calculate total number of elements to process
    int64_t total_threads = numel / dim_size;
    
    // CUDA kernel launch parameters
    const int threads = 512;
    const int blocks = (total_threads + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "cumprod_cuda_modular", ([&] {
        cumprod_kernel_modular<scalar_t><<<blocks, threads>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            numel,
            dim_size,
            stride
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &cumprod_cuda_forward_modular, "Cumulative product forward modular (CUDA)");
}