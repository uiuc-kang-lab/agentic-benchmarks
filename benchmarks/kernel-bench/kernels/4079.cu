#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void elu_kernel_optimized(const float* __restrict__ x, float* __restrict__ out, float alpha, int n) {
    int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        int pos = idx + i * blockDim.x;
        if (pos < n) {
            float val = x[pos];
            out[pos] = (val > 0) ? val : alpha * (expf(val) - 1);
        }
    }
}

torch::Tensor elu_cuda_optimized(torch::Tensor x, float alpha) {
    CHECK_INPUT(x);
    auto out = torch::empty_like(x);
    int n = x.numel();

    const int threads = 512;
    const int blocks = (n + threads * 4 - 1) / (threads * 4);

    elu_kernel_optimized<<<blocks, threads>>>(x.data_ptr<float>(), out.data_ptr<float>(), alpha, n);

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &elu_cuda_optimized, "Optimized ELU without atomics (CUDA)");
}