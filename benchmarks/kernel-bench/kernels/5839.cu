#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>

template <typename scalar_t>
__global__ void max_pool3d_forward_kernel_tuned(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t* __restrict__ indices,
    const int batch_size,
    const int channels,
    const int input_d, const int input_h, const int input_w,
    const int output_d, const int output_h, const int output_w,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation) {

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total = batch_size * channels * output_d * output_h * output_w;
    if (idx >= total) return;

    // Compute output indices
    const int w_out = idx % output_w;
    const int h_out = (idx / output_w) % output_h;
    const int d_out = (idx / (output_w * output_h)) % output_d;
    const int c = (idx / (output_w * output_h * output_d)) % channels;
    const int b = idx / (output_w * output_h * output_d * channels);

    // Compute start positions
    const int d_start = d_out * stride - padding;
    const int h_start = h_out * stride - padding;
    const int w_start = w_out * stride - padding;

    // Precompute bounds for each dimension
    const int k_d_min = (d_start < 0) ? ((-d_start + dilation - 1) / dilation) : 0;
    const int valid_d_max = (input_d - d_start + dilation - 1) / dilation;
    const int k_d_max = min(kernel_size, valid_d_max);

    const int k_h_min = (h_start < 0) ? ((-h_start + dilation - 1) / dilation) : 0;
    const int valid_h_max = (input_h - h_start + dilation - 1) / dilation;
    const int k_h_max = min(kernel_size, valid_h_max);

    const int k_w_min = (w_start < 0) ? ((-w_start + dilation - 1) / dilation) : 0;
    const int valid_w_max = (input_w - w_start + dilation - 1) / dilation;
    const int k_w_max = min(kernel_size, valid_w_max);

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    int max_index = -1;

    #pragma unroll 4
    for (int k_d = k_d_min; k_d < k_d_max; k_d++) {
        const int d_in = d_start + k_d * dilation;
        #pragma unroll 4
        for (int k_h = k_h_min; k_h < k_h_max; k_h++) {
            const int h_in = h_start + k_h * dilation;
            #pragma unroll 4
            for (int k_w = k_w_min; k_w < k_w_max; k_w++) {
                const int w_in = w_start + k_w * dilation;
                const int input_idx = (((b * channels + c) * input_d + d_in) * input_h + h_in) * input_w + w_in;
                const scalar_t val = input[input_idx];
                if (val > max_val) {
                    max_val = val;
                    max_index = input_idx;
                }
            }
        }
    }

    output[idx] = max_val;
    if (indices != nullptr) {
        indices[idx] = max_index;
    }
}

torch::Tensor max_pool3d_cuda_forward_tuned(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation,
    bool return_indices,
    bool ceil_mode) {

    auto input_sizes = input.sizes();
    const int batch_size = input_sizes[0];
    const int channels = input_sizes[1];
    const int input_d = input_sizes[2];
    const int input_h = input_sizes[3];
    const int input_w = input_sizes[4];

    const int output_d = ceil_mode ? 
        ceil((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    const int output_h = ceil_mode ?
        ceil((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    const int output_w = ceil_mode ?
        ceil((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);

    auto output = torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options());
    auto indices = return_indices ? 
        torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options().dtype(torch::kLong)) :
        torch::Tensor();

    // Use block size of 128 threads
    const int threads = 128;
    const int total = batch_size * channels * output_d * output_h * output_w;
    const int blocks = (total + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool3d_forward_cuda_tuned", ([&] {
        max_pool3d_forward_kernel_tuned<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            return_indices ? indices.data_ptr<int64_t>() : nullptr,
            batch_size, channels,
            input_d, input_h, input_w,
            output_d, output_h, output_w,
            kernel_size, stride, padding, dilation);
    }));

    if (return_indices) {
        return torch::stack({output, indices}, 0);
    }
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool3d_cuda_forward_tuned, "Max Pool 3D forward tuned (CUDA)");
}