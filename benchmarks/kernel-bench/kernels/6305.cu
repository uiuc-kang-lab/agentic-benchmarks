#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

constexpr int BLOCK_SIZE = 256;

// CUDA kernel using grid-stride loops to process all output elements
__global__ void avg_pool3d_stride_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int batch_size,
    int channels,
    int in_d, int in_h, int in_w,
    int out_d, int out_h, int out_w,
    int kernel_size, int stride, int padding) {

    int total = batch_size * channels * out_d * out_h * out_w;
    int grid_stride = blockDim.x * gridDim.x;

    // Each thread processes multiple output elements if necessary
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x;
         idx < total;
         idx += grid_stride) {

        // Decode the linear index into 5D indices: (n, c, d_out, h_out, w_out)
        int w_out = idx % out_w;
        int tmp = idx / out_w;
        int h_out = tmp % out_h;
        tmp = tmp / out_h;
        int d_out = tmp % out_d;
        tmp = tmp / out_d;
        int c = tmp % channels;
        int n = tmp / channels;

        // Compute pooling window boundaries
        int d_start = d_out * stride - padding;
        int h_start = h_out * stride - padding;
        int w_start = w_out * stride - padding;
        int d_end = d_start + kernel_size;
        int h_end = h_start + kernel_size;
        int w_end = w_start + kernel_size;

        // Clamp boundaries to valid input dimensions
        int d_start_clamped = d_start < 0 ? 0 : d_start;
        int h_start_clamped = h_start < 0 ? 0 : h_start;
        int w_start_clamped = w_start < 0 ? 0 : w_start;
        int d_end_clamped = d_end > in_d ? in_d : d_end;
        int h_end_clamped = h_end > in_h ? in_h : h_end;
        int w_end_clamped = w_end > in_w ? in_w : w_end;

        float sum = 0.0f;
        // Accumulate the sum over the pooling window
        for (int d = d_start_clamped; d < d_end_clamped; ++d) {
            for (int h = h_start_clamped; h < h_end_clamped; ++h) {
                for (int w = w_start_clamped; w < w_end_clamped; ++w) {
                    int input_index = (((n * channels + c) * in_d + d) * in_h + h) * in_w + w;
                    sum += input[input_index];
                }
            }
        }

        // For count_include_pad=True, divide by the full kernel volume
        int pool_volume = kernel_size * kernel_size * kernel_size;
        output[idx] = sum / static_cast<float>(pool_volume);
    }
}

at::Tensor forward(at::Tensor input, int kernel_size, int stride, int padding) {
    TORCH_CHECK(input.dim() == 5, "Input tensor must be 5-dimensional");
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    int batch = input.size(0);
    int channels = input.size(1);
    int in_d = input.size(2);
    int in_h = input.size(3);
    int in_w = input.size(4);

    // Calculate output dimensions
    int out_d = (in_d + 2 * padding - kernel_size) / stride + 1;
    int out_h = (in_h + 2 * padding - kernel_size) / stride + 1;
    int out_w = (in_w + 2 * padding - kernel_size) / stride + 1;

    auto output = at::empty({batch, channels, out_d, out_h, out_w}, input.options());
    int total = batch * channels * out_d * out_h * out_w;
    int threads = BLOCK_SIZE;
    int blocks = (total + threads - 1) / threads;
    if (blocks > 65535) blocks = 65535; // Enforce CUDA grid dimension limits

    avg_pool3d_stride_kernel<<<blocks, threads>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        batch, channels,
        in_d, in_h, in_w,
        out_d, out_h, out_w,
        kernel_size, stride, padding);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D Average Pooling forward using grid-stride loops");
}
