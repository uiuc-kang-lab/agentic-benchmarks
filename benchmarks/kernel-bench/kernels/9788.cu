#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

// Improved depthwise convolution kernel using grid-stride loops for better occupancy
// and minimizing the use of atomic operations. Each thread computes a unique output element,
// thus no atomic operations are required on global memory, which reduces contention.
__global__ void fast_depthwise_conv2d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch,
    int channels,
    int in_h,
    int in_w,
    int out_h,
    int out_w,
    int kernel_h,
    int stride,
    int padding,
    int dilation) {

    int total = batch * channels * out_h * out_w;
    // Grid-stride loop to cover all output elements
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < total; index += blockDim.x * gridDim.x) {
        // Decode the flattened index into (b, c, oh, ow)
        int ow = index % out_w;
        int tmp = index / out_w;
        int oh = tmp % out_h;
        tmp = tmp / out_h;
        int c = tmp % channels;
        int b = tmp / channels;

        float sum = 0.0f;
        // Precompute base offsets to reduce repeated arithmetic
        int input_channel_offset = ((b * channels) + c) * in_h * in_w;
        int weight_offset = c * kernel_h; // weight layout: (channels, 1, kernel_h, 1)
        int input_horizontal = ow * stride - padding;  // since kernel width is 1

        // Loop over kernel height dimension
        for (int kh = 0; kh < kernel_h; ++kh) {
            int ih = oh * stride - padding + kh * dilation;
            int iw = input_horizontal; 
            if (ih >= 0 && ih < in_h && iw >= 0 && iw < in_w) {
                int input_idx = input_channel_offset + ih * in_w + iw;
                int weight_idx = weight_offset + kh;
                sum += input[input_idx] * weight[weight_idx];
            }
        }
        // Add bias for the channel
        sum += bias[c];
        int output_idx = ((b * channels + c) * out_h + oh) * out_w + ow;
        output[output_idx] = sum;
    }
}

// Forward function for depthwise 2D convolution
at::Tensor forward(
    at::Tensor x,
    at::Tensor weight,
    c10::optional<at::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    // Ensure contiguous tensors
    x = x.contiguous();
    weight = weight.contiguous();

    int batch = x.size(0);
    int channels = x.size(1);
    int in_h = x.size(2);
    int in_w = x.size(3);
    int kernel_h = weight.size(2);  // weight shape: (channels, 1, kernel_h, 1)

    if (groups != channels) {
        throw std::invalid_argument("Depthwise convolution requires groups == number of input channels.");
    }

    // Handle optional bias
    at::Tensor bias_val;
    if (bias.has_value() && bias.value().defined()) {
        bias_val = bias.value().contiguous();
    } else {
        bias_val = at::zeros({channels}, x.options());
    }

    // Compute output dimensions
    int out_h = (in_h + 2 * padding - dilation * (kernel_h - 1) - 1) / stride + 1;
    int out_w = (in_w + 2 * padding - 1) / stride + 1;

    auto output = at::empty({batch, channels, out_h, out_w}, x.options());

    int total = batch * channels * out_h * out_w;
    int threads = (total < 1024) ? total : 1024;
    int blocks = (total + threads - 1) / threads;

    const float* x_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias_val.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();

    fast_depthwise_conv2d_kernel<<<blocks, threads>>>(
        x_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch,
        channels,
        in_h,
        in_w,
        out_h,
        out_w,
        kernel_h,
        stride,
        padding,
        dilation
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Fast Depthwise 2D Convolution forward (CUDA)",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = c10::nullopt,
          py::arg("stride"),
          py::arg("padding"),
          py::arg("dilation"),
          py::arg("groups"));
}
