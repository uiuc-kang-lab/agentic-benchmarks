#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void hardsigmoid_kernel(const scalar_t* __restrict__ input,
                                   scalar_t* __restrict__ output,
                                   size_t numel) {
    extern __shared__ char shared_mem[];
    scalar_t* shared_data = reinterpret_cast<scalar_t*>(shared_mem);
    
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int idx = bid * blockDim.x + tid;
    const int stride = blockDim.x * gridDim.x;
    
    // Constants for computation
    const scalar_t three = 3.0f;
    const scalar_t inv6 = 1.0f/6.0f;
    
    for (size_t i = idx; i < numel; i += stride) {
        // Load data into shared memory
        shared_data[tid] = input[i];
        __syncthreads();  // Single sync point after load
        
        // Process data from shared memory
        scalar_t x = shared_data[tid];
        scalar_t y = (x + three) * inv6;
        y = y < 0.0f ? 0.0f : (y > 1.0f ? 1.0f : y);
        
        // Write directly to global memory - no sync needed
        output[i] = y;
    }
}

torch::Tensor forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");
    auto output = torch::empty_like(input);
    const size_t numel = input.numel();
    const int threads = 256;
    const int blocks = (numel + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "hardsigmoid_cuda", ([&] {
        const int shared_mem_size = threads * sizeof(scalar_t);
        hardsigmoid_kernel<scalar_t><<<blocks, threads, shared_mem_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            numel);
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "HardSigmoid activation forward (CUDA)");
}