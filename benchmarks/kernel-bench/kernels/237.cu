#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32

__global__ void bmm_thread_block_optimized_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int batch_size,
    int M,
    int K,
    int N
) {
    int b = blockIdx.z;
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    float sum = 0.0f;
    
    const float* batch_A = A + b * M * K;
    const float* batch_B = B + b * K * N;
    
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; t++) {
        int tiledCol = t * TILE_SIZE + threadIdx.x;
        int tiledRow = t * TILE_SIZE + threadIdx.y;

        if (row < M && tiledCol < K) {
            As[threadIdx.y][threadIdx.x] = batch_A[row * K + tiledCol];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        if (tiledRow < K && col < N) {
            Bs[threadIdx.y][threadIdx.x] = batch_B[tiledRow * N + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();
        
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        
        __syncthreads();
    }
    
    if (row < M && col < N) {
        C[b * M * N + row * N + col] = sum;
    }
}

torch::Tensor forward_bmm(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 3, "A must be 3D");
    TORCH_CHECK(B.dim() == 3, "B must be 3D");
    TORCH_CHECK(A.size(0) == B.size(0), "Batch sizes must match");
    TORCH_CHECK(A.size(2) == B.size(1), "Inner dimensions (K) must match");

    int batch_size = A.size(0);
    int M = A.size(1);
    int K = A.size(2);
    int N = B.size(2);

    auto options = torch::TensorOptions().dtype(A.dtype()).device(A.device());
    auto C = torch::zeros({batch_size, M, N}, options);

    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, 
              (M + TILE_SIZE - 1) / TILE_SIZE, 
              batch_size);

    bmm_thread_block_optimized_kernel<<<grid, block>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        batch_size, M, K, N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_bmm, "Batched matrix multiplication with optimized thread and block indexing (CUDA)");
}