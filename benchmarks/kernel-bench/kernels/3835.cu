#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_softplus(scalar_t x) {
    if (x > 20.0) {
        return x;
    } else if (x < -20.0) {
        return exp(x);
    } else {
        const scalar_t exp_x = exp(x);
        return log1p(exp_x);
    }
}

template <typename scalar_t>
__global__ void softplus_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int unroll_factor = 4;
    
    // Unroll loop to process multiple elements per thread
    for (int idx = tid; idx < size; idx += stride * unroll_factor) {
        #pragma unroll
        for (int i = 0; i < unroll_factor; i++) {
            if (idx + i * stride < size) {
                const scalar_t x = input[idx + i * stride];
                output[idx + i * stride] = compute_softplus(x);
            }
        }
    }
}

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    const int threads = 256;
    const int blocks = (size + threads * 4 - 1) / (threads * 4);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "softplus_forward_cuda", ([&] {
        softplus_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}