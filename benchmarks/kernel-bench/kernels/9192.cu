#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

namespace py = pybind11;

// This kernel implements conv_transpose2d using a gather approach with stride loops.
// Each thread computes multiple output pixels by iterating over the workload in strides.

__global__ void conv_transpose2d_forward_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int N,
    int in_channels,
    int in_h,
    int in_w,
    int out_channels,
    int kernel_h,
    int kernel_w,
    int out_h,
    int out_w,
    int stride_h,
    int stride_w,
    int pad_h,
    int pad_w,
    bool has_bias
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = N * out_channels * out_h * out_w;
    int stride = blockDim.x * gridDim.x;

    for (int idx = index; idx < total; idx += stride) {
        // Decode the output index into (n, oc, out_y, out_x) assuming row-major layout
        int out_x = idx % out_w;
        int tmp = idx / out_w;
        int out_y = tmp % out_h;
        tmp = tmp / out_h;
        int oc = tmp % out_channels;
        int n = tmp / out_channels;

        float sum = has_bias ? bias[oc] : 0.0f;

        // Optimized loop ordering: iterate over kernel spatial dimensions first to minimize redundant computations
        int base_y = out_y + pad_h;
        int base_x = out_x + pad_w;
        int input_channel_stride = in_h * in_w;
        int weight_kernel_stride = kernel_h * kernel_w;
        for (int ky = 0; ky < kernel_h; ky++) {
            int t_y = base_y - ky;
            if (t_y % stride_h != 0) continue;  // Skip if not aligned
            int in_y = t_y / stride_h;
            if (in_y < 0 || in_y >= in_h) continue;
            for (int kx = 0; kx < kernel_w; kx++) {
                int t_x = base_x - kx;
                if (t_x % stride_w != 0) continue;  // Skip if not aligned
                int in_x = t_x / stride_w;
                if (in_x < 0 || in_x >= in_w) continue;
                // Precompute offsets for input and weight for this kernel location
                int input_offset = (n * in_channels) * input_channel_stride + in_y * in_w + in_x;
                int weight_offset = oc * weight_kernel_stride + ky * kernel_w + kx;
                for (int ic = 0; ic < in_channels; ic++) {
                    int input_idx = input_offset + ic * input_channel_stride;
                    int weight_idx = (ic * out_channels) * weight_kernel_stride + weight_offset;
                    sum += input[input_idx] * weight[weight_idx];
                }
            }
        }

        output[idx] = sum;
    }
}


torch::Tensor conv_transpose2d_forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding
) {
    // Input shape: [N, in_channels, in_h, in_w]
    auto N = input.size(0);
    auto in_channels = input.size(1);
    auto in_h = input.size(2);
    auto in_w = input.size(3);

    // Weight shape: [in_channels, out_channels, kernel_h, kernel_w]
    auto out_channels = weight.size(1);
    auto kernel_h = weight.size(2);
    auto kernel_w = weight.size(3);

    int stride_h = stride[0];
    int stride_w = stride[1];
    int pad_h = padding[0];
    int pad_w = padding[1];

    // Compute output dimensions based on standard transposed convolution formula
    int out_h = (in_h - 1) * stride_h - 2 * pad_h + kernel_h;
    int out_w = (in_w - 1) * stride_w - 2 * pad_w + kernel_w;

    auto output = torch::zeros({N, out_channels, out_h, out_w}, input.options());

    int total = N * out_channels * out_h * out_w;
    int threads = 256;
    int blocks = (total + threads - 1) / threads;

    bool has_bias = (bias.has_value() && bias.value().numel() > 0);
    const float* bias_ptr = has_bias ? bias.value().data_ptr<float>() : nullptr;

    conv_transpose2d_forward_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        N,
        in_channels,
        in_h,
        in_w,
        out_channels,
        kernel_h,
        kernel_w,
        out_h,
        out_w,
        stride_h,
        stride_w,
        pad_h,
        pad_w,
        has_bias
    );

    return output;
}


// Entry point from Python
torch::Tensor conv_transpose2d_forward(
    torch::Tensor input,
    torch::Tensor weight,
    py::object bias_obj,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding
) {
    c10::optional<torch::Tensor> bias = c10::nullopt;
    if (!bias_obj.is_none()) {
        bias = bias_obj.cast<torch::Tensor>();
    }
    return conv_transpose2d_forward_cuda(input, weight, bias, stride, padding);
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward, "Conv Transpose 2D forward (gather approach with stride loops)",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride"),
          py::arg("padding"));
}
