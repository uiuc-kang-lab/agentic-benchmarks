#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <iostream>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void matmul_kernel_warp(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float value = 0;
    for (int k = 0; k < K; k++) {
        value += A[row * K + k] * B[k * N + col];
    }

    // Warp-level reduction
    for (int offset = 16; offset > 0; offset /= 2) {
        value += __shfl_down_sync(0xFFFFFFFF, value, offset);
    }

    if (threadIdx.x % 32 == 0) {
        C[row * N + col] = value;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matmul_kernel_warp<<<numBlocks, threadsPerBlock>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, N, K);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication with warp-level primitives (CUDA)");
}