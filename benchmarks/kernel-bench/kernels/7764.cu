#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {
    
    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    // Use torch::conv2d with optional bias
    return torch::conv2d(
        x,
        weight,
        bias.has_value() ? bias.value() : torch::Tensor(),
        {stride, stride},
        {padding, padding},
        {dilation, dilation},
        groups
    );
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA 2D Convolution");
}