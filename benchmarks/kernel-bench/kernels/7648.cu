#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cudnn/Handles.h>
#include <ATen/cudnn/Descriptors.h>
#include <hipDNN.h>

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
#define BLOCK_SIZE_Z 4

__global__ void conv3d_shared_memory_kernel(
    float* output, const float* input, const float* weight, const float* bias,
    int batch_size, int in_channels, int out_channels,
    int in_depth, int in_height, int in_width,
    int kernel_d, int kernel_h, int kernel_w,
    int out_depth, int out_height, int out_width,
    int stride, int padding, int dilation, int groups) {

    extern __shared__ float shared_mem[];
    float* shared_input = shared_mem;
    float* shared_weight = shared_mem + BLOCK_SIZE_X * BLOCK_SIZE_Y * BLOCK_SIZE_Z;

    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;
    int tid_z = threadIdx.z;

    int w_out = blockIdx.x * blockDim.x + tid_x;
    int h_out = blockIdx.y * blockDim.y + tid_y;
    int d_out = blockIdx.z * blockDim.z + tid_z;

    if (w_out >= out_width || h_out >= out_height || d_out >= out_depth) return;

    int c_out = blockIdx.w;
    int b = blockIdx.v;

    float sum = 0.0f;

    int group = c_out / (out_channels / groups);
    int in_channels_per_group = in_channels / groups;

    for (int ic = 0; ic < in_channels_per_group; ic++) {
        int in_channel = group * in_channels_per_group + ic;

        for (int kd = 0; kd < kernel_d; kd++) {
            int d_in = d_out * stride - padding + kd * dilation;
            if (d_in < 0 || d_in >= in_depth) continue;

            for (int kh = 0; kh < kernel_h; kh++) {
                int h_in = h_out * stride - padding + kh * dilation;
                if (h_in < 0 || h_in >= in_height) continue;

                for (int kw = 0; kw < kernel_w; kw++) {
                    int w_in = w_out * stride - padding + kw * dilation;
                    if (w_in < 0 || w_in >= in_width) continue;

                    if (tid_x < kernel_w && tid_y < kernel_h && tid_z < kernel_d) {
                        shared_weight[tid_z * kernel_h * kernel_w + tid_y * kernel_w + tid_x] = 
                            weight[((c_out * in_channels_per_group + ic) * kernel_d + kd) * kernel_h * kernel_w + kh * kernel_w + kw];
                    }

                    if (tid_x < BLOCK_SIZE_X && tid_y < BLOCK_SIZE_Y && tid_z < BLOCK_SIZE_Z) {
                        shared_input[tid_z * BLOCK_SIZE_Y * BLOCK_SIZE_X + tid_y * BLOCK_SIZE_X + tid_x] = 
                            input[((b * in_channels + in_channel) * in_depth + d_in) * in_height * in_width + h_in * in_width + w_in];
                    }

                    __syncthreads();

                    sum += shared_input[tid_z * BLOCK_SIZE_Y * BLOCK_SIZE_X + tid_y * BLOCK_SIZE_X + tid_x] *
                           shared_weight[tid_z * kernel_h * kernel_w + tid_y * kernel_w + tid_x];

                    __syncthreads();
                }
            }
        }
    }

    if (bias != nullptr) {
        sum += bias[c_out];
    }

    output[(((b * out_channels + c_out) * out_depth + d_out) * out_height + h_out) * out_width + w_out] = sum;
}

at::Tensor forward(
    const at::Tensor& input,
    const at::Tensor& weight,
    const c10::optional<at::Tensor>& bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups
) {
    auto bias = bias_opt.value_or(at::Tensor());
    TORCH_CHECK(input.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "Weight must be a CUDA tensor");
    TORCH_CHECK(!bias.defined() || bias.is_cuda(), "Bias must be a CUDA tensor");

    int64_t batch_size = input.size(0);
    int64_t in_channels = input.size(1);
    int64_t in_depth = input.size(2);
    int64_t in_height = input.size(3);
    int64_t in_width = input.size(4);

    int64_t out_channels = weight.size(0);
    int64_t kernel_d = weight.size(2);
    int64_t kernel_h = weight.size(3);
    int64_t kernel_w = weight.size(4);

    int64_t out_depth = (in_depth + 2 * padding - dilation * (kernel_d - 1) - 1) / stride + 1;
    int64_t out_height = (in_height + 2 * padding - dilation * (kernel_h - 1) - 1) / stride + 1;
    int64_t out_width = (in_width + 2 * padding - dilation * (kernel_w - 1) - 1) / stride + 1;

    auto output = at::empty({batch_size, out_channels, out_depth, out_height, out_width}, input.options());

    dim3 threadsPerBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    dim3 numBlocks(
        (out_width + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X,
        (out_height + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y,
        (out_depth + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z
    );

    size_t shared_memory_size = (BLOCK_SIZE_X * BLOCK_SIZE_Y * BLOCK_SIZE_Z + kernel_d * kernel_h * kernel_w) * sizeof(float);

    conv3d_shared_memory_kernel<<<numBlocks, threadsPerBlock, shared_memory_size>>>(
        output.data_ptr<float>(),
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.defined() ? bias.data_ptr<float>() : nullptr,
        batch_size, in_channels, out_channels,
        in_depth, in_height, in_width,
        kernel_d, kernel_h, kernel_w,
        out_depth, out_height, out_width,
        stride, padding, dilation, groups
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D convolution forward with shared memory optimization (CUDA)");
}