#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void matmul_transpose_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ C,
    const int M,
    const int N,
    const int K) {
    
    constexpr int TILE_SIZE = 16;
    
    // Shared memory tiles
    __shared__ scalar_t As[TILE_SIZE][TILE_SIZE];
    __shared__ scalar_t Bs[TILE_SIZE][TILE_SIZE];
    
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int row = blockIdx.x * TILE_SIZE + tx;
    const int col = blockIdx.y * TILE_SIZE + ty;
    
    // Initialize accumulator
    scalar_t sum = 0.0f;
    
    // Loop over tiles
    for (int tile = 0; tile < (K + TILE_SIZE - 1) / TILE_SIZE; ++tile) {
        // Load tiles from global memory to shared memory
        const int tile_idx = tile * TILE_SIZE;
        
        if (row < M && (tile_idx + ty) < K) {
            // Load A - note the transposed access pattern
            As[tx][ty] = A[(tile_idx + ty) * M + row];
        } else {
            As[tx][ty] = 0.0f;
        }
        
        if (col < N && (tile_idx + tx) < K) {
            // Load B - note the transposed access pattern
            Bs[tx][ty] = B[col * K + (tile_idx + tx)];
        } else {
            Bs[tx][ty] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute partial dot product for this tile
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum = __fmaf_rn(As[tx][k], Bs[k][ty], sum);
        }
        
        __syncthreads();
    }
    
    // Write result
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

torch::Tensor matmul_transpose_cuda(torch::Tensor A, torch::Tensor B) {
    const int K = A.size(0);
    const int M = A.size(1);
    const int N = B.size(0);
    
    auto C = torch::empty({M, N}, A.options());
    
    constexpr int TILE_SIZE = 16;
    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((M + TILE_SIZE - 1) / TILE_SIZE,
                (N + TILE_SIZE - 1) / TILE_SIZE);
    
    AT_DISPATCH_FLOATING_TYPES(A.type(), "matmul_transpose_kernel", ([&] {
        matmul_transpose_kernel<scalar_t><<<blocks, threads>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, N, K
        );
    }));
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_transpose_cuda, "Matrix multiplication with transpose (CUDA)");
}