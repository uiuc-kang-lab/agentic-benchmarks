#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Device helper: define an inline exponential function for float and double.
template <typename scalar_t>
__device__ inline scalar_t my_exp(scalar_t x);

template <>
__device__ inline float my_exp<float>(float x) {
    return expf(x);
}

template <>
__device__ inline double my_exp<double>(double x) {
    return exp(x);
}

// CUDA kernel applying SELU using a grid-stride loop for improved mapping of threads to elements.
// This optimizes thread and block indexing, ensuring high occupancy and efficient execution on
// NVIDIA H100 GPUs with CUDA 12.2.

template <typename scalar_t>
__global__ void selu_kernel(const scalar_t* __restrict__ input,
                            scalar_t* __restrict__ output,
                            size_t numel) {
    // SELU parameters
    const scalar_t alpha  = static_cast<scalar_t>(1.67326324235437728481);
    const scalar_t lambda = static_cast<scalar_t>(1.05070098735548049342);

    // Grid-stride loop: each thread processes multiple elements if available.
    for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
         idx < numel;
         idx += blockDim.x * gridDim.x) {
        scalar_t x = input[idx];
        scalar_t value = (x > static_cast<scalar_t>(0)) ? x : alpha * (my_exp(x) - static_cast<scalar_t>(1));
        output[idx] = lambda * value;
    }
}

// Host function that launches the CUDA SELU kernel.
// Exposed to Python via the pybind11 module as "forward".

torch::Tensor selu_forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    auto output = torch::empty_like(input);
    size_t numel = input.numel();
    
    // Optimally setting block and grid dimensions for a grid-stride loop.
    const int threads = 1024;
    int blocks = (numel + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "selu_forward_cuda", ([&] {
        const scalar_t *input_ptr = input.data_ptr<scalar_t>();
        scalar_t *output_ptr = output.data_ptr<scalar_t>();
        selu_kernel<scalar_t><<<blocks, threads>>>(input_ptr, output_ptr, numel);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &selu_forward, "SELU Activation Forward (CUDA)");
}
