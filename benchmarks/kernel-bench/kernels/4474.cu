#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/cuda/HIPContext.h>
#include <vector>

typedef float4 float4_t;

template <typename T>
__device__ __forceinline__ T warpReduceSum(T val) {
    #pragma unroll
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

template <typename scalar_t>
__global__ void compute_stats_kernel(
    const scalar_t* __restrict__ x,
    const int N, 
    const int C,
    const int spatial,
    const int channels_per_group,
    const int num_groups,
    scalar_t* __restrict__ mean,
    scalar_t* __restrict__ var) {
    
    const int n = blockIdx.y;
    const int g = blockIdx.x;
    const int group_offset = n * C * spatial + g * channels_per_group * spatial;
    const int group_elems = channels_per_group * spatial;

    const int vec_size = sizeof(float4_t) / sizeof(scalar_t);
    const int num_vectors = group_elems / vec_size;
    const int remaining = group_elems % vec_size;
    
    scalar_t thread_sum = 0;
    scalar_t thread_sum_sq = 0;

    const float4_t* x_vec = reinterpret_cast<const float4_t*>(x + group_offset);
    for (int i = threadIdx.x; i < num_vectors; i += blockDim.x) {
        float4_t v = x_vec[i];
        thread_sum += v.x + v.y + v.z + v.w;
        thread_sum_sq += v.x * v.x + v.y * v.y + v.z * v.z + v.w * v.w;
    }

    if (threadIdx.x < remaining) {
        scalar_t val = x[group_offset + num_vectors * vec_size + threadIdx.x];
        thread_sum += val;
        thread_sum_sq += val * val;
    }

    thread_sum = warpReduceSum(thread_sum);
    thread_sum_sq = warpReduceSum(thread_sum_sq);

    __shared__ scalar_t s_partial_sums[32];
    __shared__ scalar_t s_partial_squares[32];

    const int warp_id = threadIdx.x / warpSize;
    const int lane_id = threadIdx.x % warpSize;

    if (lane_id == 0) {
        s_partial_sums[warp_id] = thread_sum;
        s_partial_squares[warp_id] = thread_sum_sq;
    }
    __syncthreads();

    if (warp_id == 0 && lane_id < (blockDim.x + warpSize - 1) / warpSize) {
        thread_sum = s_partial_sums[lane_id];
        thread_sum_sq = s_partial_squares[lane_id];
        
        thread_sum = warpReduceSum(thread_sum);
        thread_sum_sq = warpReduceSum(thread_sum_sq);

        if (lane_id == 0) {
            scalar_t group_mean = thread_sum / group_elems;
            scalar_t group_var = thread_sum_sq / group_elems - group_mean * group_mean;
            int out_index = n * num_groups + g;
            mean[out_index] = group_mean;
            var[out_index] = group_var;
        }
    }
}

template <typename scalar_t>
__global__ void group_norm_forward_kernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ mean,
    const scalar_t* __restrict__ var,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    const int N,
    const int C,
    const int spatial,
    const int channels_per_group,
    const int num_groups,
    const scalar_t eps,
    scalar_t* __restrict__ y) {

    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int total = N * C * spatial;
    
    #pragma unroll 4
    for (int idx = tid; idx < total; idx += stride) {
        const int j = idx % spatial;
        const int temp = idx / spatial;
        const int c = temp % C;
        const int n = temp / C;
        
        const int g = c / channels_per_group;
        const int stats_index = n * num_groups + g;
        
        const scalar_t m = mean[stats_index];
        const scalar_t v = var[stats_index];
        const scalar_t inv_std = rsqrt(v + eps);
        const scalar_t w = weight[c];
        const scalar_t b = bias[c];

        y[idx] = (x[idx] - m) * inv_std * w + b;
    }
}

torch::Tensor group_norm_forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    int64_t num_groups,
    double eps) {

    const int N = x.size(0);
    const int C = x.size(1);
    int spatial = 1;
    for (int i = 2; i < x.dim(); i++) {
        spatial *= x.size(i);
    }
    const int channels_per_group = C / num_groups;

    auto y = torch::empty_like(x);
    auto options = torch::TensorOptions().device(x.device()).dtype(x.dtype());
    auto mean = torch::empty({N, num_groups}, options);
    auto var = torch::empty({N, num_groups}, options);

    const int threads_stats = 256;
    dim3 blocks_stats(num_groups, N);

    const int total_elements = N * C * spatial;
    const int threads_norm = 256;
    const int blocks_norm = (total_elements + threads_norm - 1) / threads_norm;

    hipStream_t stream = c10::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "group_norm_forward_cuda", ([&] {
        compute_stats_kernel<scalar_t><<<blocks_stats, threads_stats, 0, stream>>>(
            x.data_ptr<scalar_t>(),
            N, C, spatial,
            channels_per_group,
            num_groups,
            mean.data_ptr<scalar_t>(),
            var.data_ptr<scalar_t>());

        group_norm_forward_kernel<scalar_t><<<blocks_norm, threads_norm, 0, stream>>>(
            x.data_ptr<scalar_t>(),
            mean.data_ptr<scalar_t>(),
            var.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            bias.data_ptr<scalar_t>(),
            N, C, spatial,
            channels_per_group,
            num_groups,
            static_cast<scalar_t>(eps),
            y.data_ptr<scalar_t>());
    }));

    return y;
}