#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define TILE_K 4
#define UNROLL_FACTOR 4

template <typename scalar_t>
__global__ void module_fn_cuda_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ output,
    int N, int M, int K, int L) {

    const int n = blockIdx.x;
    const int m = blockIdx.y * blockDim.y + threadIdx.y;
    const int l = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (m >= M || l >= L || n >= N) return;

    scalar_t sum = 0;
    
    // Tile K dimension
    for (int k_base = 0; k_base < K; k_base += TILE_K) {
        scalar_t tmp = 0;
        #pragma unroll
        for (int k_offset = 0; k_offset < TILE_K; k_offset += UNROLL_FACTOR) {
            int k = k_base + k_offset;
            if (k >= K) break;
            
            tmp += A[n*M*K + m*K + k] * B[k*L + l];
            if (k+1 < K) tmp += A[n*M*K + m*K + (k+1)] * B[(k+1)*L + l];
            if (k+2 < K) tmp += A[n*M*K + m*K + (k+2)] * B[(k+2)*L + l];
            if (k+3 < K) tmp += A[n*M*K + m*K + (k+3)] * B[(k+3)*L + l];
        }
        sum += tmp;
    }

    output[n*M*L + m*L + l] = sum;
}

void module_fn_cuda_forward(
    torch::Tensor A,
    torch::Tensor B,
    torch::Tensor output) {

    int N = A.size(0);
    int M = A.size(1);
    int K = A.size(2);
    int L = B.size(1);

    // 3D grid: 1 block per N, each block covers MxL
    dim3 threads(1, 16, 16);  // Total 256 threads
    dim3 blocks(
        N,
        (M + threads.y - 1) / threads.y,
        (L + threads.z - 1) / threads.z
    );

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(A.scalar_type(), "module_fn_cuda_forward", ([&] {
        module_fn_cuda_kernel<scalar_t><<<blocks, threads>>>( 
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            N, M, K, L
        );
    }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in module_fn_cuda_forward: %s\n", hipGetErrorString(err));
    }
}

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor module_fn_forward(
    torch::Tensor A,
    torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    auto N = A.size(0);
    auto M = A.size(1);
    auto L = B.size(1);

    auto output = torch::zeros({N, M, L}, A.options());
    module_fn_cuda_forward(A, B, output);
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &module_fn_forward, "module_fn forward (CUDA)");
}
