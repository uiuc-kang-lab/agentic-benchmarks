#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// CUDA kernel with stride loop optimization
// This kernel computes: output[n,m,l] = sum_{k=0}^{K-1} A[n,m,k]*B[k,l]
// where A is of shape [N, M, K] and B is of shape [K, L].

template <typename scalar_t>
__global__ void module_fn_cuda_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ output,
    int N, int M, int K, int L) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = N * M * L;
    int stride = blockDim.x * gridDim.x;

    for (int index = idx; index < total_elements; index += stride) {
        // Cache index calculations
        int n = index / (M * L);
        int m = (index % (M * L)) / L;
        int l = index % L;
        
        // Pre-calculate base offsets
        int a_base = n * M * K + m * K;
        int b_base = l;
        int out_idx = n * M * L + m * L + l;
        
        scalar_t sum = 0;
        // Main computation loop
        for (int k = 0; k < K; ++k) {
            sum += A[a_base + k] * B[b_base + k * L];
        }
        output[out_idx] = sum;
    }
}

// CUDA forward function
void module_fn_cuda_forward(
    torch::Tensor A,
    torch::Tensor B,
    torch::Tensor output) {

    int N = A.size(0);
    int M = A.size(1);
    int K = A.size(2);
    int L = B.size(1);

    const int threads = 256;
    const int blocks = (N * M * L + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(A.scalar_type(), "module_fn_cuda_forward", ([&] {
      module_fn_cuda_kernel<scalar_t><<<blocks, threads>>>(
          A.data_ptr<scalar_t>(),
          B.data_ptr<scalar_t>(),
          output.data_ptr<scalar_t>(),
          N, M, K, L);
    }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in module_fn_cuda_forward: %s\n", hipGetErrorString(err));
    }
}

// C++ interface
#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x)  TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x)  CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor module_fn_forward(
    torch::Tensor A,
    torch::Tensor B) {
  CHECK_INPUT(A);
  CHECK_INPUT(B);

  auto N = A.size(0);
  auto M = A.size(1);
  auto L = B.size(1);

  auto output = torch::zeros({N, M, L}, A.options());
  module_fn_cuda_forward(A, B, output);
  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &module_fn_forward, "module_fn forward (CUDA)");
}
