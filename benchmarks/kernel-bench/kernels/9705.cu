#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

// CUDA kernel optimizing thread and block indexing for depthwise 2D convolution
__global__ void depthwise_conv2d_thread_block_optimized(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch,
    int channels,
    int in_h,
    int in_w,
    int out_h,
    int out_w,
    int kernel_h,
    int stride,
    int padding,
    int dilation) {

    // Calculate the global thread index
    int ow = blockIdx.x * blockDim.x + threadIdx.x;
    int oh = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.z;

    if (ow < out_w && oh < out_h) {
        float sum = 0.f;
        // Loop over the vertical kernel dimension
        for (int kh = 0; kh < kernel_h; ++kh) {
            int ih = oh * stride - padding + kh * dilation;
            int iw = ow * stride - padding;  // kernel width is 1
            if (ih >= 0 && ih < in_h && iw >= 0 && iw < in_w) {
                int in_index = ((blockIdx.z * in_h + ih) * in_w + iw);
                sum += input[in_index] * weight[c * kernel_h + kh];
            }
        }
        sum += bias[c];
        int out_index = ((blockIdx.z * out_h + oh) * out_w + ow);
        output[out_index] = sum;
    }
}

// Forward function implementing depthwise convolution with optimized thread and block indexing
at::Tensor forward(
    at::Tensor x,
    at::Tensor weight,
    c10::optional<at::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    // Ensure inputs are contiguous
    x = x.contiguous();
    weight = weight.contiguous();

    int batch = x.size(0);
    int channels = x.size(1);
    int in_h = x.size(2);
    int in_w = x.size(3);
    int kernel_h = weight.size(2);  // weight shape: (channels, 1, kernel_h, 1)

    // Depthwise convolution requires groups == channels
    if (groups != channels) {
        throw std::invalid_argument("Depthwise convolution requires groups == number of input channels.");
    }

    // Handle bias: if bias is not provided, use a zeros tensor
    at::Tensor bias_val;
    if (bias.has_value() && bias.value().defined()) {
        bias_val = bias.value().contiguous();
    } else {
        bias_val = at::zeros({channels}, x.options());
    }

    // Compute output dimensions
    int out_h = (in_h + 2 * padding - dilation * (kernel_h - 1) - 1) / stride + 1;
    int out_w = (in_w + 2 * padding - 1) / stride + 1;

    auto output = at::empty({batch, channels, out_h, out_w}, x.options());

    // Define block and grid dimensions
    dim3 block(16, 16, 1);
    dim3 grid((out_w + block.x - 1) / block.x, (out_h + block.y - 1) / block.y, channels);

    const float* x_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias_val.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();

    depthwise_conv2d_thread_block_optimized<<<grid, block>>>(
        x_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch,
        channels,
        in_h,
        in_w,
        out_h,
        out_w,
        kernel_h,
        stride,
        padding,
        dilation
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Depthwise 2D Convolution forward with optimized thread and block indexing (CUDA)",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = c10::nullopt,
          py::arg("stride"),
          py::arg("padding"),
          py::arg("dilation"),
          py::arg("groups"));
}
