#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device function for computing KL divergence element
__device__ __forceinline__ float compute_kl_element(float log_pred, float target) {
    return expf(log_pred) - target * log_pred;
}

// Device function for parallel reduction
__device__ __forceinline__ void warp_reduce(volatile float* sdata, unsigned int tid) {
    if (blockDim.x >= 64) sdata[tid] += sdata[tid + 32];
    if (blockDim.x >= 32) sdata[tid] += sdata[tid + 16];
    if (blockDim.x >= 16) sdata[tid] += sdata[tid + 8];
    if (blockDim.x >= 8) sdata[tid] += sdata[tid + 4];
    if (blockDim.x >= 4) sdata[tid] += sdata[tid + 2];
    if (blockDim.x >= 2) sdata[tid] += sdata[tid + 1];
}

// Device function for block reduction
__device__ __forceinline__ void block_reduce(float* partial_sums, unsigned int tid) {
    for (int stride = blockDim.x/2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            partial_sums[tid] += partial_sums[tid + stride];
        }
        __syncthreads();
    }
    
    // Final warp reduction
    if (tid < 32) warp_reduce(partial_sums, tid);
}

// Main CUDA kernel optimized for efficient indexing
__global__ void kl_div_kernel_multi_dim(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    // Use 1D thread block indexing and 1D grid indexing
    const unsigned int tid = threadIdx.x;
    const int grid_stride = blockDim.x * gridDim.x;
    int idx = blockIdx.x * blockDim.x + tid;
    
    // Shared memory for partial sums
    extern __shared__ float partial_sums[];
    
    // Compute local sum
    float sum = 0.0f;
    for (; idx < n; idx += grid_stride) {
        sum += compute_kl_element(log_predictions[idx], targets[idx]);
    }
    
    // Store in shared memory
    partial_sums[tid] = sum;
    __syncthreads();
    
    // Perform reduction
    block_reduce(partial_sums, tid);
    
    // Write result
    if (tid == 0) {
        atomicAdd(output, partial_sums[0]);
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    const int threads = 256;
    const int blocks = min((n + threads - 1) / threads, 1024);
    const int shared_mem = threads * sizeof(float);
    
    kl_div_kernel_multi_dim<<<blocks, threads, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward (CUDA)");
}