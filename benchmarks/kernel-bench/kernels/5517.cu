#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z;
    
    const int b = z / channels;
    const int c = z % channels;

    if (x >= output_width || y >= output_height || b >= batch_size) return;

    const int input_batch_offset = b * channels * input_height * input_width;
    const int input_channel_offset = c * input_height * input_width;
    
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    #pragma unroll
    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = y * stride - padding + kh * dilation;
        if (ih >= 0 && ih < input_height) {
            const int input_h_offset = ih * input_width;
            
            #pragma unroll
            for (int kw = 0; kw < kernel_size; kw++) {
                const int iw = x * stride - padding + kw * dilation;
                if (iw >= 0 && iw < input_width) {
                    const int input_idx = input_batch_offset + 
                                        input_channel_offset + 
                                        input_h_offset + 
                                        iw;
                    max_val = max(max_val, input[input_idx]);
                }
            }
        }
    }

    const int output_idx = b * (channels * output_height * output_width) +
                          c * (output_height * output_width) +
                          y * output_width +
                          x;
    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const dim3 threads(8, 8);
    const dim3 blocks(
        (output_width + threads.x - 1) / threads.x,
        (output_height + threads.y - 1) / threads.y,
        batch_size * channels
    );

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}