#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void leaky_relu_atomic_optimized_kernel(const float* x, float* out, float negative_slope, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use atomic operations only where necessary
    if (idx < n) {
        float val = x[idx];
        out[idx] = val > 0 ? val : val * negative_slope;
    }
}

torch::Tensor leaky_relu_atomic_optimized_forward(torch::Tensor x, float negative_slope) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int n = x.numel();

    const int threads = 1024;
    const int blocks = (n + threads - 1) / threads;

    leaky_relu_atomic_optimized_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(), out.data_ptr<float>(), negative_slope, n
    );

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &leaky_relu_atomic_optimized_forward, "LeakyReLU forward optimized with atomic operations (CUDA)");
}