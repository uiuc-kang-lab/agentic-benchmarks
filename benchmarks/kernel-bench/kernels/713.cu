#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <iostream>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define TILE_WIDTH 16

// Kernel using shared memory and warp-level primitives for reduction
__global__ void OptimizedMatmulKernel(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int M, int K, int N) {
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    float cValue = 0.0f;

    __shared__ float As[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

    int numTiles = (K + TILE_WIDTH - 1) / TILE_WIDTH;
    
    for (int t = 0; t < numTiles; t++) {
        int tiledCol = t * TILE_WIDTH + threadIdx.x;
        int tiledRow = t * TILE_WIDTH + threadIdx.y;

        As[threadIdx.y][threadIdx.x] = (row < M && tiledCol < K) ? A[row * K + tiledCol] : 0.0f;
        Bs[threadIdx.y][threadIdx.x] = (tiledRow < K && col < N) ? B[tiledRow * N + col] : 0.0f;

        __syncthreads();

        // Unroll the loop for better performance
        #pragma unroll
        for (int i = 0; i < TILE_WIDTH; i++) {
            cValue += As[threadIdx.y][i] * Bs[i][threadIdx.x];
        }

        __syncthreads();
    }
    
    // Use warp-level primitives for final reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        cValue += __shfl_down_sync(0xFFFFFFFF, cValue, offset);
    }

    if (threadIdx.x % warpSize == 0) {
        atomicAdd(&C[row * N + col], cValue);
    }
}

// The forward function checks input validity, allocates the output tensor, and launches the kernel
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    auto C = torch::zeros({M, N}, A.options());

    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim((N + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);

    OptimizedMatmulKernel<<<gridDim, blockDim>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, K, N);
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized matrix multiplication with shared memory and warp-level primitives (CUDA)");
}