#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void max_reduce_kernel_optimized(
    const scalar_t* input,
    scalar_t* output,
    const int64_t outer_size,
    const int64_t dim_size,
    const int64_t inner_size
) {
    extern __shared__ scalar_t shared_data[];
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_elements = outer_size * inner_size;
    
    if (idx >= total_elements) return;
    
    const int outer_idx = idx / inner_size;
    const int inner_idx = idx % inner_size;
    
    // Calculate starting position for this thread
    const int64_t start_idx = outer_idx * dim_size * inner_size + inner_idx;
    
    // Initialize with first element
    scalar_t max_val = input[start_idx];
    
    // Reduce along dimension
    for (int i = 1; i < dim_size; i++) {
        const scalar_t val = input[start_idx + i * inner_size];
        max_val = max(max_val, val);
    }
    
    // Store the result in shared memory
    shared_data[threadIdx.x] = max_val;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shared_data[threadIdx.x] = max(shared_data[threadIdx.x], shared_data[threadIdx.x + s]);
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (threadIdx.x == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}

torch::Tensor max_reduce_cuda_forward_optimized(torch::Tensor input, int64_t dim) {
    // Handle negative dimension
    if (dim < 0) dim += input.dim();
    
    // Calculate sizes
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= input.size(i);
    }
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < input.dim(); i++) {
        inner_size *= input.size(i);
    }
    
    const int64_t dim_size = input.size(dim);
    
    // Create output tensor
    auto output_sizes = input.sizes().vec();
    output_sizes.erase(output_sizes.begin() + dim);
    auto output = torch::empty(output_sizes, input.options());
    
    // Launch kernel
    const int threads = 256;
    const int blocks = (outer_size * inner_size + threads - 1) / threads;
    const int threads = 256;
    const int blocks = (outer_size * inner_size + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "max_reduce_forward_optimized", ([&] {
        const int shared_mem_size = threads * sizeof(scalar_t);
        max_reduce_kernel_optimized<scalar_t><<<blocks, threads, shared_mem_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            dim_size,
            inner_size
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_reduce_cuda_forward_optimized, "Max reduce forward optimized (CUDA)");
}