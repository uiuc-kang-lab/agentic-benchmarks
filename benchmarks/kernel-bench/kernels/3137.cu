#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define THREADS_PER_BLOCK 256

template <int VEC_SIZE, bool ALIGNED>
__global__ void softmax_kernel(const float* __restrict__ x, float* __restrict__ y, int num_features) {
    const int batch_idx = blockIdx.x;
    const int tid = threadIdx.x;
    const int vec_tid = tid * VEC_SIZE;
    const int vec_stride = blockDim.x * VEC_SIZE;

    __shared__ float sdata[(THREADS_PER_BLOCK + 31) / 32];
    const float* x_row = x + batch_idx * num_features;
    float* y_row = y + batch_idx * num_features;

    // Vectorized max reduction
    float max_val = -INFINITY;
    if (ALIGNED) {
        #pragma unroll
        for (int i = vec_tid; i < num_features; i += vec_stride) {
            float4 vec = *reinterpret_cast<const float4*>(x_row + i);
            max_val = fmaxf(max_val, fmaxf(fmaxf(vec.x, vec.y), fmaxf(vec.z, vec.w)));
        }
    } else {
        for (int i = tid; i < num_features; i += blockDim.x) {
            max_val = fmaxf(max_val, x_row[i]);
        }
    }

    // Warp-level max reduction
    for (int offset = 16; offset > 0; offset >>= 1)
        max_val = fmaxf(max_val, __shfl_down_sync(0xffffffff, max_val, offset));

    if (tid % 32 == 0)
        sdata[tid / 32] = max_val;
    __syncthreads(); // Ensure maximum is written to shared memory

    // Final block max reduction
    if (tid < 32) {
        max_val = tid < (blockDim.x / 32) ? sdata[tid] : -INFINITY;
        for (int offset = 16; offset > 0; offset >>= 1)
            max_val = fmaxf(max_val, __shfl_down_sync(0xffffffff, max_val, offset));
        if (tid == 0)
            sdata[0] = max_val;
    }
    __syncthreads(); // Ensure final max is available for all threads
    max_val = sdata[0];

    // Vectorized sum reduction
    float sum_val = 0.0f;
    if (ALIGNED) {
        #pragma unroll
        for (int i = vec_tid; i < num_features; i += vec_stride) {
            float4 vec = *reinterpret_cast<const float4*>(x_row + i);
            float4 exp_vec = {__expf(vec.x - max_val), __expf(vec.y - max_val),
                             __expf(vec.z - max_val), __expf(vec.w - max_val)};
            *reinterpret_cast<float4*>(y_row + i) = exp_vec;
            sum_val += exp_vec.x + exp_vec.y + exp_vec.z + exp_vec.w;
        }
    } else {
        for (int i = tid; i < num_features; i += blockDim.x) {
            float exp_val = __expf(x_row[i] - max_val);
            y_row[i] = exp_val;
            sum_val += exp_val;
        }
    }

    // Warp-level sum reduction
    for (int offset = 16; offset > 0; offset >>= 1)
        sum_val += __shfl_down_sync(0xffffffff, sum_val, offset);

    if (tid % 32 == 0)
        sdata[tid / 32] = sum_val;
    __syncthreads(); // Ensure sums are written to shared memory

    // Final block sum reduction
    if (tid < 32) {
        sum_val = tid < (blockDim.x / 32) ? sdata[tid] : 0;
        for (int offset = 16; offset > 0; offset >>= 1)
            sum_val += __shfl_down_sync(0xffffffff, sum_val, offset);
        if (tid == 0)
            sdata[0] = sum_val;
    }
    __syncthreads(); // Ensure normalized factor is available for all

    // Vectorized normalization
    const float inv_sum = 1.0f / sdata[0];
    if (ALIGNED) {
        #pragma unroll
        for (int i = vec_tid; i < num_features; i += vec_stride) {
            float4 vec = *reinterpret_cast<float4*>(y_row + i);
            vec.x *= inv_sum;
            vec.y *= inv_sum;
            vec.z *= inv_sum;
            vec.w *= inv_sum;
            *reinterpret_cast<float4*>(y_row + i) = vec;
        }
    } else {
        for (int i = tid; i < num_features; i += blockDim.x)
            y_row[i] *= inv_sum;
    }
}

void softmax_forward_cuda(const float* x, float* y, int batch_size, int num_features) {
    const int vec_size = 4;
    const bool aligned = (num_features % vec_size) == 0;
    const int shared_mem = ((THREADS_PER_BLOCK + 31) / 32) * sizeof(float);

    if (aligned) {
        softmax_kernel<vec_size, true><<<batch_size, THREADS_PER_BLOCK, shared_mem>>>(x, y, num_features);
    } else {
        softmax_kernel<1, false><<<batch_size, THREADS_PER_BLOCK, shared_mem>>>(x, y, num_features);
    }
}

torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor.");
    TORCH_CHECK(x.dim() == 2, "Input tensor must be 2D.");
    TORCH_CHECK(x.scalar_type() == torch::kFloat32, "Input tensor must be float32.");

    auto y = torch::empty_like(x);
    softmax_forward_cuda(x.data_ptr<float>(), y.data_ptr<float>(), x.size(0), x.size(1));
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Softmax forward (CUDA)");
}