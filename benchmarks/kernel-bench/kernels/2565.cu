#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized CUDA kernel for ReLU activation
// Combines vectorization and block size tuning
template <typename scalar_t>
__global__ void optimized_relu_kernel(
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ input,
    const int64_t size) {

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = gridDim.x * blockDim.x;

    // Vectorized processing for aligned data
    if constexpr (sizeof(scalar_t) == 4) {
        constexpr int VEC_SIZE = 4;
        using vec_t = float4;
        const int vec_size = size / VEC_SIZE;
        const vec_t* in_vec = reinterpret_cast<const vec_t*>(input);
        vec_t* out_vec = reinterpret_cast<vec_t*>(output);

        for (int i = idx; i < vec_size; i += stride) {
            vec_t val = __ldg(&in_vec[i]);
            val.x = val.x > 0 ? val.x : 0;
            val.y = val.y > 0 ? val.y : 0;
            val.z = val.z > 0 ? val.z : 0;
            val.w = val.w > 0 ? val.w : 0;
            out_vec[i] = val;
        }

        // Handle remaining elements
        const int scalar_idx = vec_size * VEC_SIZE + idx;
        if (scalar_idx < size) {
            for (int i = scalar_idx; i < size; i += stride) {
                output[i] = __ldg(&input[i]) > 0 ? input[i] : 0;
            }
        }
    } else {
        for (int i = idx; i < size; i += stride) {
            output[i] = __ldg(&input[i]) > 0 ? input[i] : 0;
        }
    }
}

// PyTorch wrapper function
torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int64_t size = input.numel();

    // Select block size based on input size
    int threads;
    if (size > 1048576) threads = 512;
    else if (size > 10240) threads = 256;
    else threads = 128;

    const int blocks = (size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "optimized_relu_kernel", ([&] {
        optimized_relu_kernel<scalar_t><<<blocks, threads>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized ReLU forward (CUDA)");
}