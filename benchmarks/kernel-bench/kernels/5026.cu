#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void l2_normalize_kernel(
    const scalar_t* input,
    scalar_t* output,
    const int C,
    const int total_vectors,
    const int stride_C,
    const int outer_stride) {

    const int vector_idx = blockIdx.x;
    if (vector_idx >= total_vectors) return;

    const int base_offset = vector_idx * outer_stride;
    const int lane_id = threadIdx.x % 32;
    const int warp_id = threadIdx.x / 32;
    const int warps_per_block = blockDim.x / 32;

    scalar_t sum = 0.0;

    // Compute sum of squares
    for (int k = threadIdx.x; k < C; k += blockDim.x) {
        const scalar_t val = input[base_offset + k * stride_C];
        sum += val * val;
    }

    // Warp-level reduction
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    // Inter-warp reduction using first thread of each warp
    if (lane_id == 0) {
        volatile scalar_t* warp_sums = reinterpret_cast<volatile scalar_t*>(__shfl_sync(0xffffffff, sum, 0));
        if (warp_id == 0) {
            scalar_t total_sum = sum;
            #pragma unroll
            for (int w = 1; w < warps_per_block; w++) {
                total_sum += __shfl_sync(0xffffffff, warp_sums[w], w * 32);
            }
            sum = total_sum;
        }
    }

    // Broadcast final sum to all threads
    const scalar_t final_sum = __shfl_sync(0xffffffff, sum, 0);
    const scalar_t inv_norm = 1.0 / (sqrt(final_sum) + 1e-12);

    // Normalize and write output
    for (int k = threadIdx.x; k < C; k += blockDim.x) {
        output[base_offset + k * stride_C] = input[base_offset + k * stride_C] * inv_norm;
    }
}

torch::Tensor forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(input.dim() >= 1, "Input must have at least 1 dimension");

    const int C = input.size(1);
    const int total_vectors = input.numel() / C;
    const int stride_C = input.stride(1);
    const int outer_stride = input.stride(0);

    auto output = torch::empty_like(input);

    const int threads = 256;
    const int blocks = total_vectors;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "l2_normalize", [&] {
        l2_normalize_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            C,
            total_vectors,
            stride_C,
            outer_stride
        );
    });

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "L2 normalization along dim=1");
}