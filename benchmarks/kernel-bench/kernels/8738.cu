#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device function to compute the output value for a single output element
__device__ inline float compute_output_value(
    const float* __restrict__ x,       // Input tensor: [N, C_in, L_in]
    const float* __restrict__ weight,  // Weight tensor: [C_in, C_out, K_w]
    const float* __restrict__ bias,    // Bias tensor: [C_out] or nullptr
    int n, int c_out, int l_out,         // Output indices
    int C_in, int L_in, int K_w,
    int stride, int padding, int dilation) {

    float value = (bias != nullptr) ? bias[c_out] : 0.0f;

    // Loop over input channels and kernel positions
    for (int c_in = 0; c_in < C_in; ++c_in) {
        // Compute base offsets
        int x_offset = n * (C_in * L_in) + c_in * L_in;
        int w_offset = c_in * (/*C_out*/  /* note: using parameter c_out base offset */  + 0) ;
        // Actually, use the formula: w_index = c_in * (C_out*K_w) + c_out*K_w + k_w
        int weight_base = c_in * (/*C_out*K_w*/ C_out * K_w) + c_out * K_w;
        
        for (int k_w = 0; k_w < K_w; ++k_w) {
            int l_in_nom = l_out + padding - k_w * dilation;
            if (l_in_nom % stride != 0)
                continue;
            int l_in = l_in_nom / stride;
            if (l_in >= 0 && l_in < L_in) {
                value += x[x_offset + l_in] * weight[weight_base + k_w];
            }
        }
    }
    return value;
}

// Modular kernel that uses the device function to compute each output element
__global__ void conv_transpose1d_kernel_modular(
    const float* __restrict__ x,       // [N, C_in, L_in]
    const float* __restrict__ weight,  // [C_in, C_out, K_w]
    const float* __restrict__ bias,    // [C_out] or nullptr
    float* __restrict__ y,             // [N, C_out, L_out]
    int N, int C_in, int C_out, int L_in, int L_out, int K_w,
    int stride, int padding, int dilation) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = N * C_out * L_out;

    // Grid-stride loop over all output elements
    for (int index = idx; index < total_elements; index += blockDim.x * gridDim.x) {
        int l_out = index % L_out;
        int c_out = (index / L_out) % C_out;
        int n = index / (L_out * C_out);

        y[n * C_out * L_out + c_out * L_out + l_out] =
            compute_output_value(x, weight, bias,
                                 n, c_out, l_out,
                                 C_in, L_in, K_w,
                                 stride, padding, dilation);
    }
}

// Host function to set up and launch the CUDA kernel
torch::Tensor conv_transpose1d_forward(
    py::object x_obj,
    py::object weight_obj,
    py::object bias_obj = py::none(),
    int64_t stride = 1,
    int64_t padding = 0,
    int64_t dilation = 1) {

    // Convert inputs to torch::Tensor and ensure they are contiguous
    torch::Tensor x = x_obj.cast<torch::Tensor>().contiguous();
    torch::Tensor weight = weight_obj.cast<torch::Tensor>().contiguous();
    
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA device");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA device");

    float* bias_ptr = nullptr;
    if (!bias_obj.is_none()) {
        torch::Tensor bias = bias_obj.cast<torch::Tensor>().contiguous();
        TORCH_CHECK(bias.is_cuda(), "Bias tensor must be on CUDA device");
        bias_ptr = bias.data_ptr<float>();
    }

    // Get tensor dimensions
    int N = x.size(0);
    int C_in = x.size(1);
    int L_in = x.size(2);
    int K_w = weight.size(2);
    int C_out = weight.size(1);

    // Compute output length
    int L_out = (L_in - 1) * stride - 2 * padding + dilation * (K_w - 1) + 1;

    // Allocate output tensor
    auto y = torch::empty({N, C_out, L_out}, x.options());

    // Launch kernel with grid-stride loop
    int total_elements = N * C_out * L_out;
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;

    conv_transpose1d_kernel_modular<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        y.data_ptr<float>(),
        N, C_in, C_out, L_in, L_out, K_w,
        stride, padding, dilation);

    TORCH_CHECK(hipGetLastError() == hipSuccess, "CUDA kernel failed");
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        &conv_transpose1d_forward,
        "Modular Conv Transpose1D forward (CUDA)",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = 1,
        py::arg("padding") = 0,
        py::arg("dilation") = 1);
}
