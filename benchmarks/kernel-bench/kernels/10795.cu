#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void uniform_flow_cumsum_kernel(
    const scalar_t* __restrict__ x,
    const bool* __restrict__ mask,
    scalar_t* __restrict__ output,
    int64_t N,
    int64_t L) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= N) return;

    const scalar_t* x_row = x + row * L;
    const bool* mask_row = mask + row * L;
    scalar_t* output_row = output + row * L;

    scalar_t sum = 0;
    
    // Process in chunks of 32 elements (warp size) for better memory coalescing
    constexpr int CHUNK_SIZE = 32;
    scalar_t chunk_values[CHUNK_SIZE];
    
    #pragma unroll
    for (int64_t base = 0; base < L; base += CHUNK_SIZE) {
        // Pre-load chunk values and convert mask to multiplier
        #pragma unroll
        for (int i = 0; i < CHUNK_SIZE; i++) {
            int64_t idx = base + i;
            if (idx < L) {
                // Convert boolean mask to multiplier (0 or 1) to avoid branching
                scalar_t multiplier = static_cast<scalar_t>(mask_row[idx]);
                chunk_values[i] = x_row[idx] * multiplier;
            } else {
                chunk_values[i] = 0;
            }
        }

        // Compute running sum for the chunk
        #pragma unroll
        for (int i = 0; i < CHUNK_SIZE; i++) {
            int64_t idx = base + i;
            if (idx < L) {
                sum += chunk_values[i];
                output_row[idx] = sum;
            }
        }
    }
}

torch::Tensor masked_cumsum(
    const torch::Tensor& x,
    const torch::Tensor& mask,
    int64_t dim) {

    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(mask.is_cuda(), "mask must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(mask.is_contiguous(), "mask must be contiguous");
    TORCH_CHECK(x.sizes() == mask.sizes(), "x and mask must have the same shape");
    TORCH_CHECK(mask.scalar_type() == torch::kBool, "mask must be a boolean tensor");

    if (dim < 0) {
        dim += x.dim();
    }
    TORCH_CHECK(dim >= 0 && dim < x.dim(), "Invalid dimension");

    std::vector<int64_t> perm;
    for (int64_t i = 0; i < x.dim(); ++i) {
        if (i != dim)
            perm.push_back(i);
    }
    perm.push_back(dim);

    auto x_permuted = x.permute(perm).contiguous();
    auto mask_permuted = mask.permute(perm).contiguous();

    int64_t N = x_permuted.numel() / x_permuted.size(-1);
    int64_t L = x_permuted.size(-1);

    auto x_flat = x_permuted.view({N, L});
    auto mask_flat = mask_permuted.view({N, L});
    auto output_flat = torch::empty_like(x_flat);

    // Launch kernel with optimal thread configuration
    const int threads = 256;
    const int blocks = (N + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "masked_cumsum_cuda", ([&] {
        uniform_flow_cumsum_kernel<scalar_t><<<blocks, threads>>>(
            x_flat.data_ptr<scalar_t>(),
            mask_flat.data_ptr<bool>(),
            output_flat.data_ptr<scalar_t>(),
            N,
            L
        );
    }));

    auto output_permuted = output_flat.view(x_permuted.sizes());
    std::vector<int64_t> inv_perm(perm.size());
    for (size_t i = 0; i < perm.size(); ++i) {
        inv_perm[perm[i]] = i;
    }
    return output_permuted.permute(inv_perm);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &masked_cumsum, "Masked Cumulative Sum (CUDA)");
}