#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__device__ void warp_reduce_sum(scalar_t& val) {
    for (int offset = 16; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
}

template <typename scalar_t>
__global__ void triplet_margin_loss_kernel(
    const scalar_t* anchor,
    const scalar_t* positive,
    const scalar_t* negative,
    scalar_t* output,
    const float margin,
    const int batch_size,
    const int feat_size) {
    
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int warp_id = tid / 32;
    const int lane_id = tid % 32;
    
    if (warp_id >= batch_size) return;
    
    scalar_t dist_pos = 0.0;
    scalar_t dist_neg = 0.0;
    
    for (int i = lane_id; i < feat_size; i += 32) {
        const int idx = warp_id * feat_size + i;
        const scalar_t a = anchor[idx];
        const scalar_t p = positive[idx];
        const scalar_t n = negative[idx];
        
        const scalar_t d_pos = a - p;
        const scalar_t d_neg = a - n;
        
        dist_pos += d_pos * d_pos;
        dist_neg += d_neg * d_neg;
    }
    
    warp_reduce_sum(dist_pos);
    warp_reduce_sum(dist_neg);
    
    if (lane_id == 0) {
        const scalar_t loss = fmaxf(0.0, sqrtf(dist_pos) - sqrtf(dist_neg) + static_cast<scalar_t>(margin));
        output[warp_id] = loss;
    }
}

torch::Tensor triplet_margin_loss_cuda(
    torch::Tensor anchor,
    torch::Tensor positive,
    torch::Tensor negative,
    float margin) {
    
    TORCH_CHECK(anchor.device().is_cuda(), "anchor must be a CUDA tensor");
    TORCH_CHECK(positive.device().is_cuda(), "positive must be a CUDA tensor");
    TORCH_CHECK(negative.device().is_cuda(), "negative must be a CUDA tensor");
    
    const int batch_size = anchor.size(0);
    const int feat_size = anchor.size(1);
    auto output = torch::zeros({batch_size}, anchor.options());
    
    const int threads_per_block = 256;
    const int warps_per_block = threads_per_block / 32;
    const int blocks = (batch_size + warps_per_block - 1) / warps_per_block;
    
    AT_DISPATCH_FLOATING_TYPES(anchor.scalar_type(), "triplet_margin_loss_kernel", ([&] {
        triplet_margin_loss_kernel<scalar_t><<<blocks, threads_per_block>>>(
            anchor.data_ptr<scalar_t>(),
            positive.data_ptr<scalar_t>(),
            negative.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            margin,
            batch_size,
            feat_size);
    }));
    
    return output.mean();
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &triplet_margin_loss_cuda, "Triplet margin loss forward (CUDA)");
}