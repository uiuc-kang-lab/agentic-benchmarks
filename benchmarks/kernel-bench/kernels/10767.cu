#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void subtract_shifted_cumsum(const scalar_t* cumsum, scalar_t* output, int64_t dim_size, int64_t num_elements) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_elements) return;
    
    const int64_t slice = idx / dim_size;
    const int64_t pos = idx % dim_size;
    const scalar_t total = cumsum[slice * dim_size + dim_size - 1];
    
    // Use warp-shuffle to broadcast total within warp (32 threads share same slice)
    scalar_t warp_total = __shfl_sync(0xFFFFFFFF, total, (threadIdx.x % 32));
    const int64_t shifted_pos = (pos == 0) ? -1 : pos - 1;
    
    output[idx] = warp_total - (shifted_pos >= 0 ? cumsum[slice * dim_size + shifted_pos] : 0);
}

at::Tensor reverse_cumsum_optimized(at::Tensor x, int64_t dim) {
    x = x.contiguous();
    TORCH_CHECK(x.is_cuda(), "Input must be on CUDA");
    
    auto cumsum = x.cumsum(dim);
    auto output = torch::empty_like(x);
    
    const int64_t dim_size = x.size(dim);
    const int64_t num_elements = x.numel();
    
    const int threads = 256;
    const int blocks = (num_elements + threads - 1) / threads;
    
    AT_DISPATCH_ALL_TYPES(x.scalar_type(), "reverse_cumsum", [&] {
        subtract_shifted_cumsum<scalar_t><<<blocks, threads>>>(
            cumsum.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            dim_size,
            num_elements
        );
    });
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &reverse_cumsum_optimized, "Optimized reverse cumsum with warp-broadcast total");
}