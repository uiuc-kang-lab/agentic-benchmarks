#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel with grid-stride loop and a block size of 512 threads
__global__ void conv1d_kernel_opt(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int B,
    int in_channels,
    int in_size,
    int out_channels,
    int kernel_size,
    int out_size,
    int stride,
    int dilation) {

    int total_elements = B * out_channels * out_size;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_stride = blockDim.x * gridDim.x;

    // Grid-stride loop: each thread processes multiple output elements
    for (; idx < total_elements; idx += grid_stride) {
        int o = idx % out_size;
        int tmp = idx / out_size;
        int oc = tmp % out_channels;
        int b = tmp / out_channels;

        float sum = 0.0f;
        for (int ic = 0; ic < in_channels; ++ic) {
            for (int k = 0; k < kernel_size; ++k) {
                int pos = o * stride + k * dilation;
                if (pos < in_size) {
                    int x_idx = b * (in_channels * in_size) + ic * in_size + pos;
                    int w_idx = oc * (in_channels * kernel_size) + ic * kernel_size + k;
                    sum += x[x_idx] * weight[w_idx];
                }
            }
        }

        if (bias != nullptr) {
            sum += bias[oc];
        }
        
        int out_idx = b * (out_channels * out_size) + oc * out_size + o;
        output[out_idx] = sum;
    }
}

// Forward function exposed via pybind11
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int dilation) {

    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(weight.dim() == 3, "weight must be 3D");
    TORCH_CHECK(weight.size(1) == x.size(1), "Input channels mismatch");

    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        TORCH_CHECK(bias->dim() == 1, "bias must be 1D");
        TORCH_CHECK(bias->size(0) == weight.size(0), "Bias size mismatch");
    }

    int B = x.size(0);
    int in_channels = x.size(1);
    int in_size = x.size(2);
    int out_channels = weight.size(0);
    int kernel_size = weight.size(2);

    int out_size = (in_size - dilation * (kernel_size - 1) - 1) / stride + 1;
    TORCH_CHECK(out_size > 0, "Invalid output size");

    auto output = torch::empty({B, out_channels, out_size}, x.options());
    if (output.numel() == 0) return output;

    const float* x_data = x.data_ptr<float>();
    const float* weight_data = weight.data_ptr<float>();
    const float* bias_data = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float* output_data = output.data_ptr<float>();

    int total_elements = B * out_channels * out_size;
    int threads = 512;  // Experimenting with a block size of 512 threads per block
    int blocks = (total_elements + threads - 1) / threads;

    conv1d_kernel_opt<<<blocks, threads>>>(
        x_data,
        weight_data,
        bias_data,
        output_data,
        B,
        in_channels,
        in_size,
        out_channels,
        kernel_size,
        out_size,
        stride,
        dilation
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel launch error: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "1D convolution forward (CUDA) with block size 512 optimization");
}
