#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Declare constant memory for frequently accessed parameters
__constant__ int d_kernel_size;
__constant__ int d_stride;
__constant__ int d_padding;
__constant__ int d_input_length;
__constant__ int d_output_length;
__constant__ int d_in_channels;

// Kernel using constant memory for pooling parameters
__global__ void avg_pool1d_kernel_const(
    const float *input,
    float *output,
    int batch_size) {  // batch_size is passed as a kernel argument

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int channel = blockIdx.y;
    int batch = blockIdx.z;

    if (channel >= d_in_channels || batch >= batch_size) return;

    // Each thread processes multiple output indices using a grid-stride loop
    for (int o = idx; o < d_output_length; o += blockDim.x * gridDim.x) {
        float sum = 0.0f;
        for (int k = 0; k < d_kernel_size; ++k) {
            int pos_padded = o * d_stride + k;
            int pos_input = pos_padded - d_padding;
            if (pos_input >= 0 && pos_input < d_input_length) {
                int input_idx = batch * d_in_channels * d_input_length + channel * d_input_length + pos_input;
                sum += input[input_idx];
            }
        }
        int output_idx = batch * d_in_channels * d_output_length + channel * d_output_length + o;
        output[output_idx] = sum / d_kernel_size;
    }
}

// Forward function copying parameters to constant memory and launching the kernel
torch::Tensor avg_pool1d_forward_const(
    const torch::Tensor &x,
    int kernel_size,
    int stride,
    int padding) {

    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(x.dim() == 3, "x must be 3D (batch, channels, length)");

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int input_length = x.size(2);
    int output_length = (input_length + 2 * padding - kernel_size) / stride + 1;

    // Copy pooling parameters to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_size), &kernel_size, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_stride), &stride, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_padding), &padding, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_input_length), &input_length, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_output_length), &output_length, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_in_channels), &in_channels, sizeof(int));

    auto output = torch::empty({batch_size, in_channels, output_length}, x.options());

    // Configure threads and grid dimensions
    dim3 threads(256);
    dim3 grid(
        (output_length + threads.x - 1) / threads.x,
        in_channels,
        batch_size
    );

    avg_pool1d_kernel_const<<<grid, threads>>>(
        x.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &avg_pool1d_forward_const, "1D Average Pooling forward with constant memory (CUDA)");
}
