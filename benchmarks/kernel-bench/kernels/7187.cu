#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Macros for input checking
#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Tile dimensions for output block
#define TILE_H 16
#define TILE_W 16

// CUDA kernel leveraging shared memory to reduce global memory latency.
// Each block computes a tile of the output for one batch sample and one output channel.
// Within the block, threads cooperatively load a tile of the corresponding input channel into shared memory.
// The kernel then computes the convolution using the cached data and proper synchronization to avoid race conditions.

__global__ void conv2d_shared_mem_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int batch,
    const int in_channels,
    const int out_channels,
    const int in_height,
    const int in_width,
    const int out_height,
    const int out_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation) {

    // Determine indices based on grid and block dimensions
    // gridDim.x: batch, gridDim.y: output channel, gridDim.z: tiled spatial blocks
    int batch_idx = blockIdx.x;
    int out_channel = blockIdx.y;

    // Compute number of tiles in x direction
    int num_tiles_x = (out_width + TILE_W - 1) / TILE_W;
    int tile_idx = blockIdx.z;  // tile index in the 2D tiling
    int tile_row = tile_idx / num_tiles_x;
    int tile_col = tile_idx % num_tiles_x;

    // Top-left corner of the output tile
    int out_start_y = tile_row * TILE_H;
    int out_start_x = tile_col * TILE_W;

    // Each thread computes one output element within the tile
    int local_y = threadIdx.y;
    int local_x = threadIdx.x;
    int out_y = out_start_y + local_y;
    int out_x = out_start_x + local_x;

    // Compute the dimensions of the shared memory tile required
    // Shared tile height: (TILE_H - 1)*stride + (kernel_size - 1)*dilation + 1
    // Shared tile width:  (TILE_W - 1)*stride + (kernel_size - 1)*dilation + 1
    int sh_height = (TILE_H - 1) * stride + (kernel_size - 1) * dilation + 1;
    int sh_width  = (TILE_W - 1) * stride + (kernel_size - 1) * dilation + 1;

    // The input tile (to be loaded in shared memory) starts at:
    int in_tile_y = out_start_y * stride - padding;
    int in_tile_x = out_start_x * stride - padding;

    float sum = 0.0f;

    // Loop over input channels
    for (int ic = 0; ic < in_channels; ic++) {
        // Declare dynamically allocated shared memory
        extern __shared__ float shmem[];  // size should be sh_height * sh_width floats

        // Cooperatively load the required input tile for current input channel into shared memory
        int total_sh_elems = sh_height * sh_width;
        int thread_id = threadIdx.y * blockDim.x + threadIdx.x;
        for (int idx = thread_id; idx < total_sh_elems; idx += blockDim.x * blockDim.y) {
            int sh_y = idx / sh_width;
            int sh_x = idx % sh_width;
            int in_y = in_tile_y + sh_y;
            int in_x = in_tile_x + sh_x;
            float val = 0.0f;
            if (in_y >= 0 && in_y < in_height && in_x >= 0 && in_x < in_width) {
                int input_idx = batch_idx * (in_channels * in_height * in_width) +
                                ic * (in_height * in_width) +
                                in_y * in_width + in_x;
                val = input[input_idx];
            }
            shmem[sh_y * sh_width + sh_x] = val;
        }
        __syncthreads();

        // Each thread computes the convolution for its output pixel using the shared memory data
        if (out_y < out_height && out_x < out_width) {
            float accum = 0.0f;
            // For each element in the kernel window
            #pragma unroll
            for (int ky = 0; ky < kernel_size; ky++) {
                #pragma unroll
                for (int kx = 0; kx < kernel_size; kx++) {
                    int sh_y = local_y * stride + ky * dilation;
                    int sh_x = local_x * stride + kx * dilation;
                    float in_val = shmem[sh_y * sh_width + sh_x];
                    int weight_idx = ((out_channel * in_channels + ic) * kernel_size + ky) * kernel_size + kx;
                    accum += in_val * weight[weight_idx];
                }
            }
            sum += accum;
        }
        __syncthreads(); // Ensure all threads have finished using shared memory before next channel
    }

    // Write the final result to the output tensor if within bounds
    if (out_y < out_height && out_x < out_width) {
        if (bias != nullptr) {
            sum += bias[out_channel];
        }
        int out_idx = batch_idx * (out_channels * out_height * out_width) +
                      out_channel * (out_height * out_width) +
                      out_y * out_width + out_x;
        output[out_idx] = sum;
    }
}


// Host function to prepare and launch the kernel
// Falls back to torch::conv2d if groups != 1

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    TORCH_CHECK(groups == 1, "Only groups==1 is supported in conv2d_shared_mem_kernel");

    const int batch = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);

    const int out_channels = weight.size(0);
    const int kernel_size = weight.size(2);  // assuming square kernel

    // Compute output dimensions
    const int out_height = (in_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;
    const int out_width  = (in_width  + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;

    auto output = torch::zeros({batch, out_channels, out_height, out_width}, x.options());

    // Calculate tiling dimensions for output
    int num_tiles_x = (out_width + TILE_W - 1) / TILE_W;
    int num_tiles_y = (out_height + TILE_H - 1) / TILE_H;
    int total_tiles = num_tiles_x * num_tiles_y;

    // Set up grid and block dimensions
    dim3 grid(batch, out_channels, total_tiles);
    dim3 block(TILE_W, TILE_H);

    // Compute shared memory size (in bytes) needed per block
    int sh_height = (TILE_H - 1) * stride + (kernel_size - 1) * dilation + 1;
    int sh_width  = (TILE_W - 1) * stride + (kernel_size - 1) * dilation + 1;
    size_t shared_mem_size = sh_height * sh_width * sizeof(float);

    const float* input_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float* output_ptr = output.data_ptr<float>();

    conv2d_shared_mem_kernel<<<grid, block, shared_mem_size>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch,
        in_channels,
        out_channels,
        in_height,
        in_width,
        out_height,
        out_width,
        kernel_size,
        stride,
        padding,
        dilation);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Convolution forward using shared memory to reduce global memory latency");
}
