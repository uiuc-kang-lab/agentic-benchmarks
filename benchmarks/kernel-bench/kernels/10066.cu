#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>

namespace py = pybind11;

#define THREADS_PER_BLOCK 256
#define WARP_SIZE 32

// Depthwise convolution kernel with optimized memory access
template <typename scalar_t>
__global__ void depthwise_conv2d_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    int batch,
    int channels,
    int in_h, int in_w,
    int out_h, int out_w,
    int k,
    int stride,
    int padding,
    int dilation) {

  // Align thread indexing to warp size for coalesced memory access
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int lane_id = tid % WARP_SIZE;
  int total = batch * channels * out_h * out_w;
  
  if (tid >= total)
      return;

  // Decode index ensuring coalesced access within warps
  int ow = tid % out_w;
  int tmp = tid / out_w;
  int oh = tmp % out_h;
  tmp = tmp / out_h;
  int c = tmp % channels;
  int n = tmp / channels;

  scalar_t sum = 0;
  
  // Use __ldg for read-only data
  #pragma unroll
  for (int i = 0; i < k; ++i) {
    #pragma unroll
    for (int j = 0; j < k; ++j) {
      int ih = oh * stride - padding + i * dilation;
      int iw = ow * stride - padding + j * dilation;
      if (ih >= 0 && ih < in_h && iw >= 0 && iw < in_w) {
        int input_idx = n * channels * in_h * in_w + c * in_h * in_w + ih * in_w + iw;
        int weight_idx = c * k * k + i * k + j;
        sum += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
      }
    }
  }
  
  if (bias != nullptr) {
    sum += __ldg(&bias[c]);
  }
  
  output[tid] = sum;
}

// Pointwise convolution kernel with optimized memory access
template <typename scalar_t>
__global__ void pointwise_conv2d_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    int batch,
    int in_channels,
    int out_channels,
    int h,
    int w) {

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int lane_id = tid % WARP_SIZE;
  int total = batch * out_channels * h * w;
  
  if (tid >= total)
      return;

  // Decode index ensuring coalesced access
  int ow = tid % w;
  int tmp = tid / w;
  int oh = tmp % h;
  tmp = tmp / h;
  int oc = tmp % out_channels;
  int n = tmp / out_channels;

  scalar_t sum = 0;
  
  // Use vectorized loads where possible for better memory bandwidth
  #pragma unroll 4
  for (int ic = 0; ic < in_channels; ++ic) {
    int input_idx = n * in_channels * h * w + ic * h * w + oh * w + ow;
    int weight_idx = oc * in_channels + ic;
    sum += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
  }
  
  if (bias != nullptr) {
    sum += __ldg(&bias[oc]);
  }
  
  output[tid] = sum;
}

// Rest of the code remains the same as the reference implementation
torch::Tensor forward_cuda(
    const torch::Tensor& x,
    const torch::Tensor& depthwise_weight,
    const torch::Tensor& pointwise_weight,
    const torch::Tensor& depthwise_bias,
    const torch::Tensor& pointwise_bias,
    int stride,
    int padding,
    int dilation) {

  TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
  TORCH_CHECK(depthwise_weight.is_cuda(), "Depthwise weight must be a CUDA tensor");
  TORCH_CHECK(pointwise_weight.is_cuda(), "Pointwise weight must be a CUDA tensor");
  if (depthwise_bias.defined() && depthwise_bias.numel() > 0)
      TORCH_CHECK(depthwise_bias.is_cuda(), "Depthwise bias must be a CUDA tensor if provided");
  if (pointwise_bias.defined() && pointwise_bias.numel() > 0)
      TORCH_CHECK(pointwise_bias.is_cuda(), "Pointwise bias must be a CUDA tensor if provided");

  int batch = x.size(0);
  int in_channels = x.size(1);
  int in_h = x.size(2);
  int in_w = x.size(3);
  int k = depthwise_weight.size(2);
  int out_h = (in_h + 2 * padding - dilation * (k - 1) - 1) / stride + 1;
  int out_w = (in_w + 2 * padding - dilation * (k - 1) - 1) / stride + 1;

  auto depthwise_output = torch::empty({batch, in_channels, out_h, out_w}, x.options());

  int total_depthwise = batch * in_channels * out_h * out_w;
  int threads = THREADS_PER_BLOCK;
  int blocks = (total_depthwise + threads - 1) / threads;

  const void* depthwise_bias_ptr = (depthwise_bias.defined() && depthwise_bias.numel() > 0)
                                     ? depthwise_bias.data_ptr()
                                     : nullptr;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "depthwise_conv2d_cuda", ([&] {
    depthwise_conv2d_kernel<scalar_t><<<blocks, threads>>>(
        x.data_ptr<scalar_t>(),
        depthwise_weight.data_ptr<scalar_t>(),
        reinterpret_cast<const scalar_t*>(depthwise_bias_ptr),
        depthwise_output.data_ptr<scalar_t>(),
        batch,
        in_channels,
        in_h, in_w,
        out_h, out_w,
        k,
        stride,
        padding,
        dilation);
  }));

  int out_channels = pointwise_weight.size(0);
  auto output = torch::empty({batch, out_channels, out_h, out_w}, x.options());
  int total_pointwise = batch * out_channels * out_h * out_w;
  blocks = (total_pointwise + threads - 1) / threads;

  const void* pointwise_bias_ptr = (pointwise_bias.defined() && pointwise_bias.numel() > 0)
                                     ? pointwise_bias.data_ptr()
                                     : nullptr;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "pointwise_conv2d_cuda", ([&] {
    pointwise_conv2d_kernel<scalar_t><<<blocks, threads>>>(
        depthwise_output.data_ptr<scalar_t>(),
        pointwise_weight.data_ptr<scalar_t>(),
        reinterpret_cast<const scalar_t*>(pointwise_bias_ptr),
        output.data_ptr<scalar_t>(),
        batch,
        in_channels,
        out_channels,
        out_h, out_w);
  }));

  return output;
}

// Helper functions and PYBIND11_MODULE remain the same as reference implementation