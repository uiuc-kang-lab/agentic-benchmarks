#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// This kernel minimizes the use of atomic operations by leveraging shared memory
// for intra-block synchronization and only using atomic operations when necessary
// to handle race conditions. The kernel performs a reverse cumulative sum along
// the last dimension using warp-level shuffles and shared memory.

template <typename scalar_t>
__global__ void atomic_minimized_reverse_cumsum_kernel(const scalar_t* __restrict__ input,
                                                       scalar_t* __restrict__ output,
                                                       int64_t n) {
    // Each block processes one row
    int row = blockIdx.x;
    const int64_t row_offset = row * n;

    int tid = threadIdx.x;
    int lane = tid & 31;  // Lane index within the warp

    // Load element in reverse order
    scalar_t val = 0;
    if (tid < n) {
        val = input[row_offset + (n - 1 - tid)];
    }

    // Perform warp-level inclusive scan using shuffle intrinsics
    for (int offset = 1; offset < 32; offset *= 2) {
        scalar_t tmp = __shfl_up_sync(0xffffffff, val, offset);
        if (lane >= offset) {
            val += tmp;
        }
    }

    // Use shared memory to store the results of each warp's last thread
    __shared__ scalar_t warp_sums[32];
    int warp_id = tid >> 5;
    if (tid < n && (lane == 31 || tid == n - 1)) {
        warp_sums[warp_id] = val;
    }
    __syncthreads();

    // Compute warp offsets using a single thread to avoid atomic operations
    if (tid == 0) {
        scalar_t total = 0;
        for (int i = 0; i < 32; ++i) {
            scalar_t temp = warp_sums[i];
            warp_sums[i] = total;
            total += temp;
        }
    }
    __syncthreads();

    // Add the offset to each thread's value
    if (tid < n) {
        val += warp_sums[warp_id];
        output[row_offset + (n - 1 - tid)] = val;
    }
}

at::Tensor reverse_cumsum(at::Tensor x, int64_t dim) {
    // Ensure the tensor is contiguous and on CUDA
    x = x.contiguous();
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    
    const int ndim = x.dim();
    TORCH_CHECK(dim >= 0 && dim < ndim, "dim out of range");

    auto output = at::empty_like(x);

    int64_t n = x.size(dim);
    int64_t outer = x.numel() / n;

    // Use the optimized kernel only if the cum-sum is performed along the last dimension and n is not large
    if (dim == ndim - 1 && n <= 1024) {
        // Determine number of threads as next power of 2 >= n (capped at 1024)
        int threads = 1;
        while (threads < n) {
            threads *= 2;
        }
        if (threads > 1024) threads = 1024;

        dim3 blocks(outer);
        dim3 threadsPerBlock(threads);

        AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "atomic_minimized_reverse_cumsum_kernel", ([&] {
            atomic_minimized_reverse_cumsum_kernel<scalar_t><<<blocks, threadsPerBlock>>>(
                x.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                n);
        }));
    } else {
        // Fallback to the flip-cumsum-flip method for non-ideal cases
        auto x_flipped = x.flip(dim);
        auto cumsum = x_flipped.cumsum(dim);
        output = cumsum.flip(dim);
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &reverse_cumsum, "Reverse cumulative sum with minimized atomic operations (CUDA)");
}
