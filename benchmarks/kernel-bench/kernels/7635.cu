#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Constant memory declarations for frequently accessed parameters
__constant__ int c_stride[3];
__constant__ int c_padding[3];
__constant__ int c_dims[10];  // Store N, C_in, D_in, H_in, W_in, C_out, D_out, H_out, W_out, groups

// Optimized CUDA kernel with improved thread and block indexing
__global__ void conv_transposed_3d_cuda_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int kD, int kH, int kW
) {
    // Calculate 3D grid and block indices
    int w = blockIdx.x * blockDim.x + threadIdx.x;
    int h = blockIdx.y * blockDim.y + threadIdx.y;
    int d = blockIdx.z * blockDim.z + threadIdx.z;

    if (w >= c_dims[8] || h >= c_dims[7] || d >= c_dims[6]) return; // Bounds check

    for (int n = 0; n < c_dims[0]; ++n) {
        for (int c_out = 0; c_out < c_dims[5]; ++c_out) {
            int output_channels_per_group = c_dims[5] / c_dims[9];  // C_out / groups
            int group = c_out / output_channels_per_group;
            int c_out_in_group = c_out - group * output_channels_per_group;
            int input_channels_per_group = c_dims[1] / c_dims[9];  // C_in / groups

            float out_val = (bias != nullptr) ? bias[c_out] : 0.0f;

            for (int r = 0; r < kD; r++) {
                int d_in_calc = d + c_padding[0] - r;
                if (d_in_calc < 0 || d_in_calc % c_stride[0] != 0) continue;
                int d_in = d_in_calc / c_stride[0];
                if (d_in < 0 || d_in >= c_dims[2]) continue;

                for (int s = 0; s < kH; s++) {
                    int h_in_calc = h + c_padding[1] - s;
                    if (h_in_calc % c_stride[1] != 0) continue;
                    int h_in = h_in_calc / c_stride[1];
                    if (h_in < 0 || h_in >= c_dims[3]) continue;

                    for (int t = 0; t < kW; t++) {
                        int w_in_calc = w + c_padding[2] - t;
                        if (w_in_calc % c_stride[2] != 0) continue;
                        int w_in = w_in_calc / c_stride[2];
                        if (w_in < 0 || w_in >= c_dims[4]) continue;

                        for (int c = 0; c < input_channels_per_group; c++) {
                            int actual_c_in = group * input_channels_per_group + c;

                            int input_index = (((n * c_dims[1] + actual_c_in) * c_dims[2] + d_in) 
                                             * c_dims[3] + h_in) * c_dims[4] + w_in;
                            float in_val = input[input_index];

                            int weight_index = ((actual_c_in * output_channels_per_group + c_out_in_group) 
                                              * (kD * kH * kW)) + (r * kH * kW + s * kW + t);
                            float w_val = weight[weight_index];

                            out_val += in_val * w_val;
                        }
                    }
                }
            }

            int output_index = (((n * c_dims[5] + c_out) * c_dims[6] + d) 
                               * c_dims[7] + h) * c_dims[8] + w;
            output[output_index] = out_val;
        }
    }
}

torch::Tensor forward(
    torch::Tensor input,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> output_padding,
    int64_t groups
) {
    // Get dimensions
    const int N = input.size(0);
    const int C_in = input.size(1);
    const int D_in = input.size(2);
    const int H_in = input.size(3);
    const int W_in = input.size(4);
    const int kD = weight.size(2);
    const int kH = weight.size(3);
    const int kW = weight.size(4);
    
    // Calculate output dimensions
    const int output_channels_per_group = weight.size(1);
    const int C_out = output_channels_per_group * groups;
    const int D_out = (D_in - 1) * stride[0] - 2 * padding[0] + kD + output_padding[0];
    const int H_out = (H_in - 1) * stride[1] - 2 * padding[1] + kH + output_padding[1];
    const int W_out = (W_in - 1) * stride[2] - 2 * padding[2] + kW + output_padding[2];

    // Copy constant parameters to device
    int h_stride[3] = {static_cast<int>(stride[0]), static_cast<int>(stride[1]), static_cast<int>(stride[2])};
    int h_padding[3] = {static_cast<int>(padding[0]), static_cast<int>(padding[1]), static_cast<int>(padding[2])};
    int h_dims[10] = {N, C_in, D_in, H_in, W_in, C_out, D_out, H_out, W_out, static_cast<int>(groups)};
    
    hipMemcpyToSymbol(HIP_SYMBOL(c_stride), h_stride, sizeof(int) * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(c_padding), h_padding, sizeof(int) * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(c_dims), h_dims, sizeof(int) * 10);

    // Create output tensor
    auto output = torch::zeros({N, C_out, D_out, H_out, W_out}, input.options());

    // Launch configuration
    dim3 blockSize(8, 8, 8);  // Experiment with different block sizes for 3D grid
    dim3 gridSize((W_out + blockSize.x - 1) / blockSize.x, 
                  (H_out + blockSize.y - 1) / blockSize.y, 
                  (D_out + blockSize.z - 1) / blockSize.z);

    // Get raw pointers
    const float *input_ptr = input.data_ptr<float>();
    const float *weight_ptr = weight.data_ptr<float>();
    const float *bias_ptr = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float *output_ptr = output.data_ptr<float>();

    // Launch kernel
    conv_transposed_3d_cuda_kernel<<<gridSize, blockSize>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        kD, kH, kW
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ConvTranspose3d forward with optimized thread and block indexing",
          py::arg("input"),
          py::arg("weight"),
          py::arg("bias") = nullptr,
          py::arg("stride"),
          py::arg("padding"),
          py::arg("output_padding"),
          py::arg("groups"));
}