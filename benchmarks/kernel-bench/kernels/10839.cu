#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Modular device function for performing masked cumulative sum for one row
template <typename scalar_t>
__device__ inline void compute_masked_cumsum(const scalar_t* __restrict__ x_row,
                                              const bool* __restrict__ mask_row,
                                              scalar_t* __restrict__ output_row,
                                              int64_t L) {
    scalar_t sum = scalar_t(0);
    for (int64_t i = 0; i < L; ++i) {
        if (mask_row[i]) {
            sum += x_row[i];
        }
        output_row[i] = sum;
    }
}

// Optimized kernel that uses a modular device function for each row
template <typename scalar_t>
__global__ void optimized_masked_cumsum_kernel(
    const scalar_t* __restrict__ x,
    const bool* __restrict__ mask,
    scalar_t* __restrict__ output,
    int64_t N,
    int64_t L) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    // Process one row using the modular device function
    compute_masked_cumsum<scalar_t>(x + idx * L,
                                      mask + idx * L,
                                      output + idx * L,
                                      L);
}

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Host function that prepares tensors, launches the kernel, and reconstructs the output
torch::Tensor optimized_masked_cumsum(
    const torch::Tensor& x,
    const torch::Tensor& mask,
    int64_t dim) {

    CHECK_INPUT(x);
    CHECK_INPUT(mask);
    TORCH_CHECK(x.sizes() == mask.sizes(), "x and mask must have the same shape");
    TORCH_CHECK(mask.scalar_type() == torch::kBool, "mask must be a boolean tensor");

    if (dim < 0) {
        dim += x.dim();
    }
    TORCH_CHECK(dim >= 0 && dim < x.dim(), "Invalid dimension");

    // Permute dimensions to move the target dimension to the last
    std::vector<int64_t> perm;
    for (int64_t i = 0; i < x.dim(); ++i) {
        if (i != dim) {
            perm.push_back(i);
        }
    }
    perm.push_back(dim);

    auto x_permuted = x.permute(perm).contiguous();
    auto mask_permuted = mask.permute(perm).contiguous();

    // Reshape into a 2D tensor (N rows x L columns)
    int64_t N = x_permuted.numel() / x_permuted.size(-1);
    int64_t L = x_permuted.size(-1);

    auto x_flat = x_permuted.view({N, L});
    auto mask_flat = mask_permuted.view({N, L});
    auto output_flat = torch::empty_like(x_flat);

    // Launch the CUDA kernel
    const int threads = 256;
    const int blocks = (N + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "optimized_masked_cumsum_kernel", ([&] {
        optimized_masked_cumsum_kernel<scalar_t><<<blocks, threads>>>(
            x_flat.data_ptr<scalar_t>(),
            mask_flat.data_ptr<bool>(),
            output_flat.data_ptr<scalar_t>(),
            N,
            L);
    }));

    // Reshape and permute back to the original shape
    auto output_permuted = output_flat.view(x_permuted.sizes());
    std::vector<int64_t> inv_perm(perm.size());
    for (size_t i = 0; i < perm.size(); ++i) {
        inv_perm[perm[i]] = i;
    }
    auto output = output_permuted.permute(inv_perm);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &optimized_masked_cumsum, "Optimized Modular Masked Cumulative Sum (CUDA)");
}
