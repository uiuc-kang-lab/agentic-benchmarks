#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void softsign_kernel_coalesced(const float* __restrict__ x, float* __restrict__ out, int num_elements) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    // Single grid-stride pattern without loop to reduce potential warp divergence
    if (tid < num_elements) {
        float val = x[tid];
        out[tid] = val / (1.0f + fabsf(val));
    }
}

torch::Tensor forward(torch::Tensor x) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int num_elements = x.numel();
    int threads = 1024;
    int blocks = (num_elements + threads - 1) / threads;

    softsign_kernel_coalesced<<<blocks, threads>>>(
        x.data_ptr<float>(), out.data_ptr<float>(), num_elements
    );

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Softsign activation with coalesced memory (CUDA)");
}
