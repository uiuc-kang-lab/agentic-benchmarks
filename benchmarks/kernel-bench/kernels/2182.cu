#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

// Kernel for A.T * B utilizing warp-level reduction with __shfl_down_sync
__global__ void warpOptimizedReductionKernel(const float* __restrict__ A,
                               const float* __restrict__ B,
                               float* __restrict__ C,
                               int K,
                               int M, 
                               int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M && col < N) {
        float sum = 0.0f;
        int lane = threadIdx.x % 32;  // Warp lane index
        for (int k = 0; k < K; k += 32) {
            float partialSum = 0.0f;
            if (k + lane < K)
                partialSum = A[(k + lane) * M + row] * B[(k + lane) * N + col];

            // Warp-level reduction
            for (int offset = 16; offset > 0; offset /= 2) {
                partialSum += __shfl_down_sync(0xffffffff, partialSum, offset);
            }
            sum += partialSum;
        }

        if (lane == 0) {
            C[row * N + col] = sum;
        }
    }
}

// The forward function exposed via PyBind11.
// Inputs:
//   A: Tensor of shape (K, M) [CUDA, float32]
//   B: Tensor of shape (K, N) [CUDA, float32]
// Returns:
//   C: Tensor of shape (M, N) computed as A.T * B.
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    // Ensure inputs are CUDA tensors and of type float32.
    TORCH_CHECK(A.is_cuda(), "Input A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input B must be a CUDA tensor");
    TORCH_CHECK(A.dtype() == torch::kFloat32, "Input A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "Input B must be float32");

    int K = A.size(0);
    int M = A.size(1);
    TORCH_CHECK(B.size(0) == K, "Dimension mismatch: A and B must have the same first dimension (K)");
    int N = B.size(1);

    // Allocate output tensor C of shape (M, N).
    auto C = torch::zeros({M, N}, torch::device(A.device()).dtype(A.dtype()));

    dim3 blockDim(32, 16);
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x,
                 (N + blockDim.y - 1) / blockDim.y);

    const float* A_ptr = A.data_ptr<float>();
    const float* B_ptr = B.data_ptr<float>();
    float* C_ptr = C.data_ptr<float>();

    warpOptimizedReductionKernel<<<gridDim, blockDim>>>(A_ptr, B_ptr, C_ptr, K, M, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Compute C = A.T * B with warp-level reduction (CUDA)");
}
