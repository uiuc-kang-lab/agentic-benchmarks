#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Optimized kernel using shared memory efficiently and minimizing syncthreads
__global__ void cosine_similarity_loss_kernel_optimized(const float* __restrict__ predictions,
                                                        const float* __restrict__ targets,
                                                        float* output,
                                                        int N,
                                                        int D) {
    extern __shared__ float shared_mem[];

    int row = blockIdx.x;
    int tid = threadIdx.x;
    int blockSize = blockDim.x;

    float* shared_preds = shared_mem;
    float* shared_targets = shared_preds + D;

    // Load data into shared memory only once per row
    if (tid < D) {
        shared_preds[tid] = predictions[row * D + tid];
        shared_targets[tid] = targets[row * D + tid];
    }
    __syncthreads();

    float sum_dot = 0.0f;
    float sum_pred_sq = 0.0f;
    float sum_target_sq = 0.0f;

    for (int i = tid; i < D; i += blockSize) {
        float p = shared_preds[i];
        float t = shared_targets[i];
        sum_dot += p * t;
        sum_pred_sq += p * p;
        sum_target_sq += t * t;
    }

    // Warp-level reduction using __shfl_down_sync
    unsigned int mask = 0xffffffff;
    for (int offset = 16; offset > 0; offset /= 2) {
        sum_dot += __shfl_down_sync(mask, sum_dot, offset);
        sum_pred_sq += __shfl_down_sync(mask, sum_pred_sq, offset);
        sum_target_sq += __shfl_down_sync(mask, sum_target_sq, offset);
    }

    int lane = tid & 31;        // tid % 32
    int warpId = tid >> 5;      // tid / 32
    int numWarps = (blockSize + warpSize - 1) / warpSize;

    extern __shared__ float shared[];  // size: 3 * numWarps floats
    float* s_dot      = shared;
    float* s_pred_sq  = s_dot + numWarps;
    float* s_target_sq= s_pred_sq + numWarps;

    if (lane == 0) {
        s_dot[warpId] = sum_dot;
        s_pred_sq[warpId] = sum_pred_sq;
        s_target_sq[warpId] = sum_target_sq;
    }
    __syncthreads(); // Only synchronize here once for warp-level reduction

    if (tid < numWarps) {
        sum_dot      = s_dot[tid];
        sum_pred_sq  = s_pred_sq[tid];
        sum_target_sq= s_target_sq[tid];

        for (int offset = (numWarps >> 1); offset > 0; offset /= 2) {
            sum_dot += __shfl_down_sync(0xffffffff, sum_dot, offset);
            sum_pred_sq += __shfl_down_sync(0xffffffff, sum_pred_sq, offset);
            sum_target_sq += __shfl_down_sync(0xffffffff, sum_target_sq, offset);
        }

        if (tid == 0) {
            const float eps = 1e-8f;
            float norm_pred = sqrtf(sum_pred_sq);
            float norm_target = sqrtf(sum_target_sq);
            float denominator = norm_pred * norm_target;
            denominator = fmaxf(denominator, eps);
            float cos_sim = sum_dot / denominator;
            atomicAdd(output, 1.0f - cos_sim);
        }
    }
}

// Host function that wraps the CUDA kernel
// This function performs necessary input checks and kernel launch

torch::Tensor cosine_similarity_loss_forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.dim() == 2, "predictions must be 2D");
    TORCH_CHECK(targets.dim() == 2, "targets must be 2D");
    TORCH_CHECK(predictions.sizes() == targets.sizes(), "Input tensors must have the same shape");
    TORCH_CHECK(predictions.scalar_type() == torch::kFloat32, "predictions must be float32");
    TORCH_CHECK(targets.scalar_type() == torch::kFloat32, "targets must be float32");

    int N = predictions.size(0);
    int D = predictions.size(1);

    auto output = torch::zeros({1}, predictions.options());
    const int block_size = 256;
    size_t shared_mem = (2 * D + 3 * ((block_size + 31) / 32)) * sizeof(float); // memory for shared_preds, shared_targets, and warp sums

    // Launch one block per sample
    cosine_similarity_loss_kernel_optimized<<<N, block_size, shared_mem>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        N,
        D
    );

    output.div_(N);
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &cosine_similarity_loss_forward, "Cosine Similarity Loss Forward with optimized shared memory (CUDA)");
}
