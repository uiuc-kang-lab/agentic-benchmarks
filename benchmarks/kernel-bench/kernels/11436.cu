#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel for KL divergence calculation using stride loops
__global__ void kl_div_kernel_stride_loops(
    const float* log_predictions,
    const float* targets, 
    float* output,
    const int n) {
    
    // Get global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Shared memory for partial sums
    extern __shared__ float partial_sums[];
    
    partial_sums[threadIdx.x] = 0.0f;
    __syncthreads();
    
    // Calculate KL divergence for this thread's elements using stride loops
    for (int i = idx; i < n; i += stride) {
        float log_pred = log_predictions[i];
        float target = targets[i];
        partial_sums[threadIdx.x] += expf(log_pred) - target * log_pred;
    }
    __syncthreads();
    
    // Parallel reduction in shared memory
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            partial_sums[threadIdx.x] += partial_sums[threadIdx.x + stride];
        }
        __syncthreads();
    }
    
    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        atomicAdd(output, partial_sums[0]);
    }
}

torch::Tensor kl_div_cuda_forward_stride_loops(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    // Get tensor sizes
    const int n = log_predictions.numel();
    
    // Create output tensor
    auto output = torch::zeros({1}, log_predictions.options());
    
    // Launch parameters
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;
    const int shared_mem = threads * sizeof(float);
    
    // Launch kernel
    kl_div_kernel_stride_loops<<<blocks, threads, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward_stride_loops, "KL divergence forward using stride loops (CUDA)");
}