#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void conv_transpose1d_kernel_atomic(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ y,
    int N, int C_in, int C_out, int L_in, int L_out, int K_w,
    int stride, int padding, int dilation) {

    int l_out = blockIdx.x * blockDim.x + threadIdx.x;
    int c_out = blockIdx.y * blockDim.y + threadIdx.y;
    int n = blockIdx.z;

    if (l_out >= L_out || c_out >= C_out) return;

    float value = (bias != nullptr) ? bias[c_out] : 0.0f;

    for (int c_in = 0; c_in < C_in; ++c_in) {
        int x_base = n * C_in * L_in + c_in * L_in;
        int w_base = c_in * C_out * K_w + c_out * K_w;

        for (int k_w = 0; k_w < K_w; ++k_w) {
            int l_in_nom = l_out + padding - k_w * dilation;
            if (l_in_nom % stride != 0) continue;
            int l_in = l_in_nom / stride;
            if (l_in >= 0 && l_in < L_in) {
                float x_val = x[x_base + l_in];
                float w_val = weight[w_base + k_w];
                value += x_val * w_val;
            }
        }
    }

    atomicAdd(&y[n * C_out * L_out + c_out * L_out + l_out], value);
}

torch::Tensor conv_transpose1d_forward(
    py::object x_obj,
    py::object weight_obj,
    py::object bias_obj = py::none(),
    int64_t stride = 1,
    int64_t padding = 0,
    int64_t dilation = 1) {

    torch::Tensor x = x_obj.cast<torch::Tensor>().contiguous();
    torch::Tensor weight = weight_obj.cast<torch::Tensor>().contiguous();

    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA device");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA device");

    float* bias_ptr = nullptr;
    if (!bias_obj.is_none()) {
        torch::Tensor bias = bias_obj.cast<torch::Tensor>().contiguous();
        TORCH_CHECK(bias.is_cuda(), "Bias tensor must be on CUDA device");
        bias_ptr = bias.data_ptr<float>();
    }

    int N = x.size(0);
    int C_in = x.size(1);
    int L_in = x.size(2);
    int K_w = weight.size(2);
    int C_out = weight.size(1);
    int L_out = (L_in - 1) * stride - 2 * padding + dilation * (K_w - 1) + 1;

    auto y = torch::zeros({N, C_out, L_out}, x.options());

    dim3 threads(16, 16);
    dim3 blocks((L_out + threads.x - 1) / threads.x, (C_out + threads.y - 1) / threads.y, N);

    conv_transpose1d_kernel_atomic<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        y.data_ptr<float>(),
        N, C_in, C_out, L_in, L_out, K_w,
        stride, padding, dilation);

    TORCH_CHECK(hipGetLastError() == hipSuccess, "CUDA kernel failed");
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        &conv_transpose1d_forward,
        "Optimized Conv Transpose1D with atomic operations",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = 1,
        py::arg("padding") = 0,
        py::arg("dilation") = 1);
}
