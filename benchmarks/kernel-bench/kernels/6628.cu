#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized kernel using __ldg for read-only memory access and aligning memory accesses

template <typename scalar_t>
__global__ void ldg_max_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t inner_size,
    const int64_t dim_size
) {
    // Determine which outer index this block is working on
    int outer_idx = blockIdx.x;
    
    // Determine the tile index in the inner dimension
    int inner_idx = blockIdx.y * blockDim.x + threadIdx.x;
    if (inner_idx >= inner_size) return;

    // Base offset for this outer index
    int64_t base_offset = outer_idx * dim_size * inner_size;

    // Initialize maximum value with the first element in the reduction dimension
    scalar_t max_val = __ldg(&input[base_offset + inner_idx]);

    // Loop over the reduction dimension
    for (int i = 1; i < dim_size; i++) {
        scalar_t val = __ldg(&input[base_offset + i * inner_size + inner_idx]);
        max_val = max(max_val, val);
    }

    // Write the result to output
    output[outer_idx * inner_size + inner_idx] = max_val;
}

// Function to launch the optimized kernel
torch::Tensor ldg_max_reduce_cuda_forward(torch::Tensor input, int64_t dim) {
    // Handle negative dimension
    if (dim < 0) dim += input.dim();

    // Calculate sizes
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= input.size(i);
    }
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < input.dim(); i++) {
        inner_size *= input.size(i);
    }
    
    const int64_t dim_size = input.size(dim);

    // Create output tensor
    auto output_sizes = input.sizes().vec();
    output_sizes.erase(output_sizes.begin() + dim);
    auto output = torch::empty(output_sizes, input.options());

    // Configure block and grid sizes
    const int threads = 256;
    int blocks_y = (inner_size + threads - 1) / threads;
    dim3 grid(outer_size, blocks_y);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "ldg_max_reduce_forward", ([&] {
        ldg_max_reduce_kernel<scalar_t><<<grid, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            inner_size,
            dim_size
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &ldg_max_reduce_cuda_forward, "LDG Max reduction forward (CUDA)");
}