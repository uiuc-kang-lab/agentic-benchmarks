#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <cmath>

namespace py = pybind11;

// Tiling parameters for spatial dimensions and output channel chunks
#define TILE_SIZE 16
#define TILE_OC 8  // tile size for output channels to limit register usage

// Fused kernel that computes depthwise and pointwise convolution in one shot
// For each output pixel (n, oh, ow), we loop over input channels 'c'.
// For each input channel, we compute the depthwise convolution result (dw) using the kernel (k x k).
// Then, we multiply by the corresponding pointwise weights and accumulate into output channel accumulators.

template <typename scalar_t>
__global__ void fused_conv2d_kernel(
    const scalar_t* __restrict__ input,             // [batch, in_channels, in_h, in_w]
    const scalar_t* __restrict__ depthwise_weight,    // [in_channels, 1, k, k]
    const scalar_t* __restrict__ pointwise_weight,    // [out_channels, in_channels]
    const scalar_t* __restrict__ depthwise_bias,      // [in_channels] or nullptr
    const scalar_t* __restrict__ pointwise_bias,      // [out_channels] or nullptr
    scalar_t* __restrict__ output,                    // [batch, out_channels, out_h, out_w]
    int batch,
    int in_channels,
    int out_channels,
    int in_h, int in_w,
    int out_h, int out_w,
    int k,
    int stride,
    int padding,
    int dilation) {

  // Map the thread to an output spatial coordinate and batch index
  int n = blockIdx.z;  // one block layer per image in batch
  int ow = blockIdx.x * blockDim.x + threadIdx.x;
  int oh = blockIdx.y * blockDim.y + threadIdx.y;

  if (ow < out_w && oh < out_h) {
    // Process output channels in tiles to limit per-thread register usage
    for (int oc_tile = 0; oc_tile < out_channels; oc_tile += TILE_OC) {

      // Local accumulator for a tile of output channels
      // Initialize accumulators to zero
      scalar_t acc[TILE_OC];
      #pragma unroll
      for (int t = 0; t < TILE_OC; t++) {
        int oc = oc_tile + t;
        if (oc < out_channels)
          acc[t] = 0;
      }

      // Loop over input channels, each of which has its own depthwise kernel
      for (int c = 0; c < in_channels; c++) {
        scalar_t dw = 0;
        // Apply the depthwise convolution kernel
        for (int i = 0; i < k; i++) {
          for (int j = 0; j < k; j++) {
            int ih = oh * stride - padding + i * dilation;
            int iw = ow * stride - padding + j * dilation;
            if (ih >= 0 && ih < in_h && iw >= 0 && iw < in_w) {
              int input_idx = n * (in_channels * in_h * in_w) +
                              c * (in_h * in_w) +
                              ih * in_w + iw;
              int weight_dw_idx = c * (k * k) + i * k + j;
              dw += input[input_idx] * depthwise_weight[weight_dw_idx];
            }
          }
        }
        // Add depthwise bias if provided
        if (depthwise_bias != nullptr) {
          dw += depthwise_bias[c];
        }

        // For the current input channel result, update the pointwise accumulators for the tile
        #pragma unroll
        for (int t = 0; t < TILE_OC; t++) {
          int oc = oc_tile + t;
          if (oc < out_channels) {
            int weight_pw_idx = oc * in_channels + c;  // pointwise weight layout: [out_channels, in_channels]
            acc[t] += dw * pointwise_weight[weight_pw_idx];
          }
        }
      }  // end loop over input channels

      // Write the accumulated results to the output tensor, adding pointwise bias if available
      for (int t = 0; t < TILE_OC; t++) {
        int oc = oc_tile + t;
        if (oc < out_channels) {
          if (pointwise_bias != nullptr)
            acc[t] += pointwise_bias[oc];
          int output_idx = n * (out_channels * out_h * out_w) +
                           oc * (out_h * out_w) +
                           oh * out_w + ow;
          output[output_idx] = acc[t];
        }
      }
    }  // end loop over output channel tiles
  }
}


// Fused forward function that sets up grid and launches the fused kernel

torch::Tensor fused_forward_cuda(
    const torch::Tensor& x,
    const torch::Tensor& depthwise_weight,
    const torch::Tensor& pointwise_weight,
    const torch::Tensor& depthwise_bias,
    const torch::Tensor& pointwise_bias,
    int stride,
    int padding,
    int dilation) {

  TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
  TORCH_CHECK(depthwise_weight.is_cuda(), "Depthwise weight must be a CUDA tensor");
  TORCH_CHECK(pointwise_weight.is_cuda(), "Pointwise weight must be a CUDA tensor");
  if (depthwise_bias.defined() && depthwise_bias.numel() > 0)
    TORCH_CHECK(depthwise_bias.is_cuda(), "Depthwise bias must be a CUDA tensor if provided");
  if (pointwise_bias.defined() && pointwise_bias.numel() > 0)
    TORCH_CHECK(pointwise_bias.is_cuda(), "Pointwise bias must be a CUDA tensor if provided");

  int batch = x.size(0);
  int in_channels = x.size(1);
  int in_h = x.size(2);
  int in_w = x.size(3);

  // Depthwise weight is of shape: [in_channels, 1, k, k]
  int k = depthwise_weight.size(2);
  int out_h = (in_h + 2 * padding - dilation * (k - 1) - 1) / stride + 1;
  int out_w = (in_w + 2 * padding - dilation * (k - 1) - 1) / stride + 1;

  // Pointwise weight shape: [out_channels, in_channels]
  int out_channels = pointwise_weight.size(0);

  auto output = torch::empty({batch, out_channels, out_h, out_w}, x.options());

  // Configure a 3D grid: grid.x and grid.y cover spatial (out_w, out_h), grid.z covers the batch dimension
  dim3 block(TILE_SIZE, TILE_SIZE);
  dim3 grid((out_w + TILE_SIZE - 1) / TILE_SIZE,
            (out_h + TILE_SIZE - 1) / TILE_SIZE,
            batch);

  const void* depthwise_bias_ptr = (depthwise_bias.defined() && depthwise_bias.numel() > 0) ? depthwise_bias.data_ptr() : nullptr;
  const void* pointwise_bias_ptr = (pointwise_bias.defined() && pointwise_bias.numel() > 0) ? pointwise_bias.data_ptr() : nullptr;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "fused_conv2d_cuda", ([&] {
    fused_conv2d_kernel<scalar_t><<<grid, block>>>(
        x.data_ptr<scalar_t>(),
        depthwise_weight.data_ptr<scalar_t>(),
        pointwise_weight.data_ptr<scalar_t>(),
        reinterpret_cast<const scalar_t*>(depthwise_bias_ptr),
        reinterpret_cast<const scalar_t*>(pointwise_bias_ptr),
        output.data_ptr<scalar_t>(),
        batch,
        in_channels,
        out_channels,
        in_h, in_w,
        out_h, out_w,
        k,
        stride,
        padding,
        dilation);
  }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Fused kernel launch error: %s\n", hipGetErrorString(err));
  }

  return output;
}

// Helper to convert a py::object to a Torch tensor. Supports tensors or objects with a 'data' attribute.

at::Tensor toTensor(const py::object& obj) {
  if (obj.is_none()) {
    return at::Tensor();
  }
  try {
    return obj.cast<at::Tensor>();
  } catch (const py::cast_error& e) {
    if (py::hasattr(obj, "data")) {
      return obj.attr("data").cast<at::Tensor>();
    }
    throw std::runtime_error("Expected a torch Tensor or Parameter.");
  }
}

// Wrapper exposed to Python. It supports inputs that may be wrapped (e.g., in nn.Parameter).

at::Tensor fused_forward_wrapper(py::object x_obj,
                                 py::object depthwise_weight_obj,
                                 py::object pointwise_weight_obj,
                                 py::object depthwise_bias_obj,
                                 py::object pointwise_bias_obj,
                                 int stride,
                                 int padding,
                                 int dilation) {
  auto x = toTensor(x_obj);
  auto depthwise_weight = toTensor(depthwise_weight_obj);
  auto pointwise_weight = toTensor(pointwise_weight_obj);
  auto depthwise_bias = toTensor(depthwise_bias_obj);
  auto pointwise_bias = toTensor(pointwise_bias_obj);

  return fused_forward_cuda(x, depthwise_weight, pointwise_weight,
                            depthwise_bias, pointwise_bias,
                            stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &fused_forward_wrapper, "Fused CUDA depthwise separable convolution (depthwise + pointwise) with a single kernel launch");
}
