#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
#define WARP_SIZE 32

__device__ __forceinline__ float warp_reduce_sum(float val) {
    #pragma unroll
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__global__ void matmul_kernel(const float* __restrict__ A,
                              const float* __restrict__ B,
                              float* __restrict__ C,
                              int M, int N, int K,
                              int lda, int ldb, int ldc,
                              bool transA, bool transB) {
    const int warpId = threadIdx.x / WARP_SIZE;
    const int laneId = threadIdx.x % WARP_SIZE;
    
    const int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    const int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    float sum = 0.0f;

    for (int t = 0; t < (K + BLOCK_SIZE - 1) / BLOCK_SIZE; ++t) {
        if (row < M && t * BLOCK_SIZE + threadIdx.x < K) {
            As[threadIdx.y][threadIdx.x] = transA ? 
                A[(t * BLOCK_SIZE + threadIdx.x) * lda + row] :
                A[row * lda + t * BLOCK_SIZE + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && t * BLOCK_SIZE + threadIdx.y < K) {
            Bs[threadIdx.y][threadIdx.x] = transB ?
                B[col * ldb + t * BLOCK_SIZE + threadIdx.y] :
                B[(t * BLOCK_SIZE + threadIdx.y) * ldb + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        float local_sum = 0.0f;
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            local_sum += As[threadIdx.y][k + laneId % (WARP_SIZE/4)] * 
                        Bs[k + laneId % (WARP_SIZE/4)][threadIdx.x];
        }

        sum += warp_reduce_sum(local_sum);

        __syncthreads();
    }

    if (row < M && col < N) {
        if (laneId == 0) {
            C[row * ldc + col] = sum;
        }
    }
}

torch::Tensor matmul_cuda(torch::Tensor A, torch::Tensor B) {
    if (!A.is_cuda() || !B.is_cuda()) {
        throw std::invalid_argument("Input tensors must be on CUDA devices");
    }
    if (A.dim() != 2 || B.dim() != 2) {
        throw std::invalid_argument("Input tensors must be 2D matrices");
    }

    int64_t A_rows = A.size(0);
    int64_t A_cols = A.size(1);
    int64_t B_rows = B.size(0);
    int64_t B_cols = B.size(1);

    bool transA = false;
    bool transB = false;
    int64_t M, N, K;
    int lda, ldb, ldc;

    if (A_rows >= A_cols && B_rows == A_cols) {
        M = A_rows; K = A_cols; N = B_cols;
        lda = A.stride(0); ldb = B.stride(0);
    } else if (A_cols > A_rows && B_rows == A_rows) {
        transA = true;
        M = A_cols; K = A_rows; N = B_cols;
        lda = A.stride(1); ldb = B.stride(0);
    } else if (A_rows >= A_cols && B_cols == A_cols) {
        transB = true;
        M = A_rows; K = A_cols; N = B_rows;
        lda = A.stride(0); ldb = B.stride(1);
    } else if (A_cols > A_rows && B_cols == A_rows) {
        transA = true; transB = true;
        M = A_cols; K = A_rows; N = B_rows;
        lda = A.stride(1); ldb = B.stride(1);
    } else {
        throw std::invalid_argument("Incompatible matrix dimensions");
    }

    ldc = N;
    auto C = torch::empty({M, N}, A.options());

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
                 (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matmul_kernel<<<gridDim, blockDim>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        M, N, K,
        lda, ldb, ldc,
        transA, transB);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_cuda, "Warp-shuffle matrix multiplication (CUDA)");
}