#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>

namespace py = pybind11;

#define THREADS_PER_BLOCK 256
#define ELEMENTS_PER_THREAD 4

template <typename scalar_t>
__global__ void depthwise_conv2d_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    int batch,
    int channels,
    int in_h, int in_w,
    int out_h, int out_w,
    int k,
    int stride,
    int padding,
    int dilation) {

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = blockDim.x * gridDim.x;
  int total = batch * channels * out_h * out_w;

  // Process multiple elements per thread
  for (int index = tid; index < total; index += stride_x) {
    int ow = index % out_w;
    int tmp = index / out_w;
    int oh = tmp % out_h;
    tmp = tmp / out_h;
    int c = tmp % channels;
    int n = tmp / channels;

    scalar_t sum = 0;
    #pragma unroll
    for (int i = 0; i < k; ++i) {
      #pragma unroll
      for (int j = 0; j < k; ++j) {
        int ih = oh * stride - padding + i * dilation;
        int iw = ow * stride - padding + j * dilation;
        if (ih >= 0 && ih < in_h && iw >= 0 && iw < in_w) {
          int input_idx = n * channels * in_h * in_w + c * in_h * in_w + ih * in_w + iw;
          int weight_idx = c * k * k + i * k + j;
          sum += input[input_idx] * weight[weight_idx];
        }
      }
    }
    if (bias != nullptr)
      sum += bias[c];
    output[index] = sum;
  }
}

// Shared memory tile size for pointwise convolution
#define TILE_DIM 32

template <typename scalar_t>
__global__ void pointwise_conv2d_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    int batch,
    int in_channels,
    int out_channels,
    int h,
    int w) {

  __shared__ scalar_t shared_input[TILE_DIM][TILE_DIM + 1];  // +1 to avoid bank conflicts

  int bx = blockIdx.x * TILE_DIM;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int h_idx = bx + tx;
  int batch_channel = by;
  int n = batch_channel / out_channels;
  int oc = batch_channel % out_channels;

  scalar_t sum = 0;
  
  // Loop over input channel tiles
  for (int tile = 0; tile < (in_channels + TILE_DIM - 1) / TILE_DIM; ++tile) {
    // Load input tile into shared memory
    int ic = tile * TILE_DIM + ty;
    if (h_idx < h * w && ic < in_channels) {
      shared_input[ty][tx] = input[n * in_channels * h * w + ic * h * w + h_idx];
    } else {
      shared_input[ty][tx] = 0;
    }
    __syncthreads();

    // Compute partial sums
    #pragma unroll
    for (int k = 0; k < TILE_DIM && (tile * TILE_DIM + k) < in_channels; ++k) {
      sum += shared_input[k][tx] * weight[oc * in_channels + tile * TILE_DIM + k];
    }
    __syncthreads();
  }

  if (h_idx < h * w) {
    if (bias != nullptr)
      sum += bias[oc];
    output[n * out_channels * h * w + oc * h * w + h_idx] = sum;
  }
}

torch::Tensor forward_cuda(
    const torch::Tensor& x,
    const torch::Tensor& depthwise_weight,
    const torch::Tensor& pointwise_weight,
    const torch::Tensor& depthwise_bias,
    const torch::Tensor& pointwise_bias,
    int stride,
    int padding,
    int dilation) {

  TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");

  int batch = x.size(0);
  int in_channels = x.size(1);
  int in_h = x.size(2);
  int in_w = x.size(3);
  int k = depthwise_weight.size(2);
  int out_h = (in_h + 2 * padding - dilation * (k - 1) - 1) / stride + 1;
  int out_w = (in_w + 2 * padding - dilation * (k - 1) - 1) / stride + 1;

  auto depthwise_output = torch::empty({batch, in_channels, out_h, out_w}, x.options());

  int total_depthwise = batch * in_channels * out_h * out_w;
  int threads = THREADS_PER_BLOCK;
  int blocks = (total_depthwise + threads * ELEMENTS_PER_THREAD - 1) / (threads * ELEMENTS_PER_THREAD);

  const void* depthwise_bias_ptr = (depthwise_bias.defined() && depthwise_bias.numel() > 0)
                                     ? depthwise_bias.data_ptr()
                                     : nullptr;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "depthwise_conv2d_cuda", ([&] {
    depthwise_conv2d_kernel<scalar_t><<<blocks, threads>>>(
        x.data_ptr<scalar_t>(),
        depthwise_weight.data_ptr<scalar_t>(),
        reinterpret_cast<const scalar_t*>(depthwise_bias_ptr),
        depthwise_output.data_ptr<scalar_t>(),
        batch,
        in_channels,
        in_h, in_w,
        out_h, out_w,
        k,
        stride,
        padding,
        dilation);
  }));

  int out_channels = pointwise_weight.size(0);
  auto output = torch::empty({batch, out_channels, out_h, out_w}, x.options());

  dim3 threadsPoint(TILE_DIM, TILE_DIM);
  dim3 blocksPoint((out_h * out_w + TILE_DIM - 1) / TILE_DIM,
                   batch * out_channels);

  const void* pointwise_bias_ptr = (pointwise_bias.defined() && pointwise_bias.numel() > 0)
                                     ? pointwise_bias.data_ptr()
                                     : nullptr;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "pointwise_conv2d_cuda", ([&] {
    pointwise_conv2d_kernel<scalar_t><<<blocksPoint, threadsPoint>>>(
        depthwise_output.data_ptr<scalar_t>(),
        pointwise_weight.data_ptr<scalar_t>(),
        reinterpret_cast<const scalar_t*>(pointwise_bias_ptr),
        output.data_ptr<scalar_t>(),
        batch,
        in_channels,
        out_channels,
        out_h, out_w);
  }));

  return output;
}

at::Tensor toTensor(const py::object& obj) {
  if (obj.is_none()) {
    return at::Tensor();
  }
  try {
    return obj.cast<at::Tensor>();
  } catch (const py::cast_error& e) {
    if (py::hasattr(obj, "data")) {
      return obj.attr("data").cast<at::Tensor>();
    }
    throw std::runtime_error("Expected a torch Tensor or Parameter.");
  }
}

at::Tensor forward_wrapper(py::object x_obj,
                           py::object depthwise_weight_obj,
                           py::object pointwise_weight_obj,
                           py::object depthwise_bias_obj,
                           py::object pointwise_bias_obj,
                           int stride,
                           int padding,
                           int dilation) {

  auto x = toTensor(x_obj);
  auto depthwise_weight = toTensor(depthwise_weight_obj);
  auto pointwise_weight = toTensor(pointwise_weight_obj);
  auto depthwise_bias = toTensor(depthwise_bias_obj);
  auto pointwise_bias = toTensor(pointwise_bias_obj);

  return forward_cuda(x, depthwise_weight, pointwise_weight,
                      depthwise_bias, pointwise_bias,
                      stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward_wrapper, "CUDA depthwise separable convolution forward");
}