#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   const int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    const int bx = blockIdx.x * TILE_SIZE;
    const int by = blockIdx.y * TILE_SIZE;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    const int row = by + ty;
    const int col = bx + tx;
    
    float sum = 0.0f;
    
    // Only compute for lower triangular portion
    if (row >= col && row < N && col < N) {
        // Loop over tiles
        for (int t = 0; t < N; t += TILE_SIZE) {
            // Collaborative loading of tiles into shared memory
            if (row < N && (t + tx) < N)
                As[ty][tx] = A[row * N + (t + tx)];
            else
                As[ty][tx] = 0.0f;
            
            if ((t + ty) < N && col < N)
                Bs[ty][tx] = B[(t + ty) * N + col];
            else
                Bs[ty][tx] = 0.0f;
            
            __syncthreads();
            
            // Compute partial dot product for this tile
            // Only accumulate for valid k indices (k >= col && k <= row)
            for (int k = 0; k < TILE_SIZE; k++) {
                int global_k = t + k;
                if (global_k >= col && global_k <= row && global_k < N) {
                    sum += As[ty][k] * Bs[k][tx];
                }
            }
            
            __syncthreads();
        }
        
        // Write result
        C[row * N + col] = sum;
    } else if (row < col && row < N && col < N) {
        // Set upper triangular portion to zero
        C[row * N + col] = 0.0f;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, 
              (N + TILE_SIZE - 1) / TILE_SIZE);

    triangular_mm_kernel<<<grid, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}