#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

__global__ void einsum_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int BATCH, int I, int J, int L, int K
) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_idx >= BATCH * I * J * K) return;

    int k = global_idx % K;
    int remainder = global_idx / K;
    int j = remainder % J;
    remainder /= J;
    int i = remainder % I;
    int b = remainder / I;

    float sum = 0.0f;
    for(int l = 0; l < L; ++l) {
        int a_offset = b * I*J*L + i*J*L + j*L + l;
        int b_offset = l*K + k;
        sum += A[a_offset] * B[b_offset];
    }
    
    C[global_idx] = sum;
}

void forward_with_streams(torch::Tensor A, torch::Tensor B, torch::Tensor C, hipStream_t stream) {
    int BATCH = A.size(0), I = A.size(1), J = A.size(2), L = A.size(3);
    int K = B.size(1);
    
    int total_elements = BATCH * I * J * K;
    
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;
    
    einsum_kernel<<<blocks, threads, 0, stream>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        BATCH, I, J, L, K
    );
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(A.dim() == 4, "A must be 4D");
    TORCH_CHECK(B.dim() == 2, "B must be 2D");
    TORCH_CHECK(A.size(3) == B.size(0), "Dimension mismatch in l");

    int BATCH = A.size(0), I = A.size(1), J = A.size(2), L = A.size(3);
    int K = B.size(1);
    
    auto C = torch::zeros({BATCH, I, J, K}, A.options());
    
    hipStream_t stream;
    hipStreamCreate(&stream);

    forward_with_streams(A, B, C, stream);

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "4D tensor-matrix multiplication with streams (CUDA)");
}