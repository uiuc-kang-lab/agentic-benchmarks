#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32

#define CHECK_INPUT(x) \
    TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor"); \
    TORCH_CHECK(x.is_contiguous(), #x " must be contiguous"); \
    TORCH_CHECK(x.scalar_type() == torch::kFloat, #x " must be a float tensor");

__global__ void optimizedMatrixMultiplyKernel(const float* __restrict__ A,
                                               const float* __restrict__ B,
                                               float* __restrict__ C,
                                               int M, int N, int K) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;
    int numTiles = (K + TILE_SIZE - 1) / TILE_SIZE;
    for (int t = 0; t < numTiles; t++) {
        if (row < M && t * TILE_SIZE + threadIdx.x < K) {
            As[threadIdx.y][threadIdx.x] = __ldg(&A[row * K + t * TILE_SIZE + threadIdx.x]);
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && t * TILE_SIZE + threadIdx.y < K) {
            Bs[threadIdx.y][threadIdx.x] = __ldg(&B[(t * TILE_SIZE + threadIdx.y) * N + col]);
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

void matrix_multiply_cuda(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);

    optimizedMatrixMultiplyKernel<<<blocks, threads>>>(A.data_ptr<float>(),
                                                       B.data_ptr<float>(),
                                                       C.data_ptr<float>(),
                                                       M, N, K);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess){
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());
    matrix_multiply_cuda(A, B, C);
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Matrix Multiplication (CUDA)");
}