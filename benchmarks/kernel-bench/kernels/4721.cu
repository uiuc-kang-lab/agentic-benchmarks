#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// CUDA kernel for computing sum of squares with minimized synchronizations
__global__ void compute_norm_kernel_min_sync(const float* input, float* norm_out, int numel) {
    __shared__ float sdata[256];
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;
    float sum = 0.0f;

    // Each thread computes its partial sum
    while (idx < numel) {
        float val = input[idx];
        sum += val * val;
        idx += blockDim.x * gridDim.x;
    }

    // Write partial sum to shared memory
    sdata[tid] = sum;
    __syncthreads(); // Necessary to ensure all partial sums are in shared memory

    // First reduction stage: combine 256 -> 128
    if (tid < 128) {
        sdata[tid] += sdata[tid + 128];
    }
    __syncthreads(); // Wait for first stage to complete

    // Second stage: reduce 128 -> 64 using shared memory
    if (tid < 64) {
        volatile float* vsdata = sdata; // Volatile to avoid extra syncs
        vsdata[tid] += vsdata[tid + 64];
        float val = vsdata[tid];

        // Warp-level reduction using shfl_down_sync
        for (int offset = 32; offset > 0; offset /= 2) {
            val += __shfl_down_sync(0xffffffff, val, offset);
        }

        // Only one thread per block writes the block's result
        if (tid == 0) {
            atomicAdd(norm_out, val);
        }
    }
}

// CUDA kernel for tensor normalization
__global__ void normalize_kernel(const float* input, float* output, float norm, int numel) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numel) {
        output[idx] = input[idx] / norm;
    }
}

// Host function interfacing with PyTorch, now using streams
torch::Tensor forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be on GPU");
    TORCH_CHECK(input.is_contiguous(), "Input tensor must be contiguous");
    TORCH_CHECK(input.scalar_type() == torch::kFloat32, "Input must be float32");

    auto output = torch::empty_like(input);
    auto norm_tensor = torch::zeros({1}, input.options());

    const float* input_ptr = input.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();
    float* norm_ptr = norm_tensor.data_ptr<float>();
    int numel = input.numel();

    const int threads = 256;
    const int blocks = min(65535, (numel + threads - 1) / threads);

    // Create CUDA streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // Launch kernel to compute the sum of squares with minimal synchronizations on stream1
    compute_norm_kernel_min_sync<<<blocks, threads, 0, stream1>>>(input_ptr, norm_ptr, numel);
    
    // Retrieve the computed sum using hipMemcpyAsync on stream1
    float norm_val;
    hipMemcpyAsync(&norm_val, norm_ptr, sizeof(float), hipMemcpyDeviceToHost, stream1);

    // Wait for the first stream to be completed before launching normalization
    hipStreamSynchronize(stream1);

    // Compute the Frobenius norm
    norm_val = sqrt(norm_val);

    // Launch kernel to normalize the tensor on stream2
    normalize_kernel<<<blocks, threads, 0, stream2>>>(input_ptr, output_ptr, norm_val, numel);

    // Wait for all operations to complete
    hipStreamSynchronize(stream2);

    // Destroy streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Frobenius norm normalization with stream overlap");
}