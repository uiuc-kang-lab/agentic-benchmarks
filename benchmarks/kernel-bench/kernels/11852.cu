#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void kldiv_atomic_optimized_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets,
    float* __restrict__ output,
    const int n) {

    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int gridSize = blockDim.x * gridDim.x;

    float sum = 0.0f;

    // Loop over the data, stride by grid size
    while (idx < n) {
        float log_pred = log_predictions[idx];
        float target = targets[idx];
        sum += expf(log_pred) - target * log_pred;
        idx += gridSize;
    }

    // Each thread writes its local sum into shared memory
    sdata[tid] = sum;
    __syncthreads();

    // Reduce within block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Only one thread per block writes to global output
    if (tid == 0) {
        atomicAdd(output, sdata[0]);
    }
}

torch::Tensor kl_div_cuda_forward(
    const torch::Tensor& log_predictions,
    const torch::Tensor& targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());

    const unsigned int threads = 256;
    const unsigned int blocks = (n + threads - 1) / threads;
    const size_t shared_mem = threads * sizeof(float);

    kldiv_atomic_optimized_kernel<<<blocks, threads, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence atomic optimized forward");
}