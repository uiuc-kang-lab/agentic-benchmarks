#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAException.h>

// Define tile size and maximum constant memory elements for matrix B
#define TILE_SIZE 32
// Maximum allowed elements in constant memory for matrix B (supports up to 128x128 matrices)
#define MAX_CONST_ELEMENTS 16384

// Declare constant memory for matrix B
__constant__ float constB[MAX_CONST_ELEMENTS];

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CHECK_FLOAT(x) TORCH_CHECK(x.scalar_type() == torch::kFloat32, #x " must be a float32 tensor")

// CUDA kernel using constant memory for matrix B
__global__ void matmul_constant_kernel(const float* __restrict__ A, float* __restrict__ C, int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;
    float C_val = 0.0f;

    int numTiles = (N + TILE_SIZE - 1) / TILE_SIZE;
    for (int m = 0; m < numTiles; ++m) {
        int tiledACol = m * TILE_SIZE + tx;
        if (row < N && tiledACol < N) {
            As[ty][tx] = A[row * N + tiledACol];
        } else {
            As[ty][tx] = 0.0f;
        }

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            int tiledBRow = m * TILE_SIZE + k;
            float B_val = (tiledBRow < N && col < N) ? constB[tiledBRow * N + col] : 0.0f;
            C_val += As[ty][k] * B_val;
        }

        __syncthreads();
    }

    if (row < N && col < N)
        C[row * N + col] = C_val;
}

// Forward function: copies matrix B into constant memory and launches the kernel
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_FLOAT(A);
    CHECK_FLOAT(B);

    TORCH_CHECK(A.dim() == 2 && A.size(0) == A.size(1), "A must be a square matrix");
    TORCH_CHECK(B.dim() == 2 && B.size(0) == B.size(1), "B must be a square matrix");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be of the same size");

    int64_t N = A.size(0);
    TORCH_CHECK(N * N <= MAX_CONST_ELEMENTS, "Matrix size too large for constant memory optimization (max N = 128)");

    auto C = torch::zeros({N, N}, A.options());

    const float* A_data = A.data_ptr<float>();
    const float* B_data = B.data_ptr<float>();
    float* C_data = C.data_ptr<float>();

    // Copy matrix B into constant memory
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(constB), B_data, N * N * sizeof(float), 0, hipMemcpyDeviceToDevice);
    C10_CUDA_CHECK(err);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    matmul_constant_kernel<<<blocksPerGrid, threadsPerBlock>>>(A_data, C_data, N);
    C10_CUDA_CHECK(hipGetLastError());

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication kernel using constant memory for matrix B (CUDA)");
}
