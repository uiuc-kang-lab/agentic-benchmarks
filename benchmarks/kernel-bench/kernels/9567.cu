#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// Maximum kernel size (assumed to be appropriate)
#define MAX_KERNEL_SIZE 16

// Optimized CUDA kernel for 2D transposed convolution with improved thread and block mappings.
// Precomputes valid kernel positions for dimensions to minimize warp divergence.
__global__ void conv_transpose2d_forward_kernel_thread_block_map(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

    // Calculate global coordinates for batch index
    int b = blockIdx.z;
    int o = blockIdx.y % out_channels;
    int w_out = blockIdx.x * blockDim.x + threadIdx.x;
    int h_out = blockIdx.y * blockDim.y + threadIdx.y;

    float out_val = (w_out < out_width && h_out < out_height) ? __ldg(&bias[o]) : 0.0f;

    if (w_out >= out_width || h_out >= out_height) return;

    for (int c = 0; c < in_channels; ++c) {
        int base_h = h_out + padding;
        int base_w = w_out + padding;

        int valid_p_count = 0;
        int valid_p[MAX_KERNEL_SIZE];
        int h_in_list[MAX_KERNEL_SIZE];
        for (int p = 0; p < kernel_size; p++) {
            int p_dilated = p * dilation;
            if (base_h >= p_dilated && ((base_h - p_dilated) % stride) == 0) {
                int h_in = (base_h - p_dilated) / stride;
                if (h_in < in_height) {
                    valid_p[valid_p_count] = p;
                    h_in_list[valid_p_count] = h_in;
                    valid_p_count++;
                }
            }
        }

        int valid_q_count = 0;
        int valid_q[MAX_KERNEL_SIZE];
        int w_in_list[MAX_KERNEL_SIZE];
        for (int q = 0; q < kernel_size; q++) {
            int q_dilated = q * dilation;
            if (base_w >= q_dilated && ((base_w - q_dilated) % stride) == 0) {
                int w_in = (base_w - q_dilated) / stride;
                if (w_in < in_width) {
                    valid_q[valid_q_count] = q;
                    w_in_list[valid_q_count] = w_in;
                    valid_q_count++;
                }
            }
        }

        // Iterate using precomputed valid indices
        for (int i = 0; i < valid_p_count; i++) {
            int p = valid_p[i];
            int h_in = h_in_list[i];
            for (int j = 0; j < valid_q_count; j++) {
                int q = valid_q[j];
                int w_in = w_in_list[j];

                int input_idx = (((b * in_channels + c) * in_height) + h_in) * in_width + w_in;
                int weight_idx = (((c * out_channels + o) * kernel_size + p) * kernel_size) + q;

                out_val += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
            }
        }
    }

    int output_idx = (((b * out_channels) + o) * out_height + h_out) * out_width + w_out;
    output[output_idx] = out_val;
}

// CUDA forward function implementation with updated block and thread mapping
torch::Tensor conv_transpose2d_forward_cuda_thread_block_map(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {

    // Get shape information
    const int batch_size = input.size(0);
    const int in_channels = input.size(1);
    const int in_height = input.size(2);
    const int in_width = input.size(3);

    const int out_channels = weight.size(1);
    const int kernel_size = weight.size(2);

    const int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
    const int out_width  = (in_width - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;

    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());

    dim3 threads(16, 16);
    dim3 blocks((out_width + 16 - 1) / 16, (out_height + 16 - 1) / 16, batch_size);

    conv_transpose2d_forward_kernel_thread_block_map<<<blocks, threads>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.data_ptr<float>(),
        output.data_ptr<float>(),
        in_channels,
        out_channels,
        in_height,
        in_width,
        kernel_size,
        out_height,
        out_width,
        stride,
        padding,
        dilation);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in conv_transpose2d_forward_kernel_thread_block_map: %s\n", hipGetErrorString(err));
    }

    return output;
}

// Wrapper function with potential bias handling
torch::Tensor conv_transpose2d_forward_wrapper_thread_block_map(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,
    int stride,
    int padding,
    int dilation) {

    int out_channels = weight.size(1);
    torch::Tensor bias;
    if (bias_obj.is(pybind11::none())) {
        bias = torch::zeros({out_channels}, weight.options());
    } else {
        bias = bias_obj.cast<torch::Tensor>();
    }
    return conv_transpose2d_forward_cuda_thread_block_map(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward_wrapper_thread_block_map,
          "ConvTranspose2d forward (CUDA) with improved thread and block mappings",
          pybind11::arg("input"),
          pybind11::arg("weight"),
          pybind11::arg("bias"),
          pybind11::arg("stride"),
          pybind11::arg("padding"),
          pybind11::arg("dilation"));
}
