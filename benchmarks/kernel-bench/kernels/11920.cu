#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void triplet_margin_loss_kernel(
    const scalar_t* __restrict__ anchor,
    const scalar_t* __restrict__ positive,
    const scalar_t* __restrict__ negative,
    scalar_t* __restrict__ output,
    const float margin,
    const int batch_size,
    const int feat_size) {
    
    __shared__ scalar_t shared_mem[32];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int sample_idx = bid;
    
    if (sample_idx >= batch_size) return;
    
    // Process features in a coalesced manner
    scalar_t dist_pos = 0.0f;
    scalar_t dist_neg = 0.0f;
    
    const int base_idx = sample_idx * feat_size;
    
    // Stride through features with multiple threads for coalesced access
    for (int feat_idx = tid; feat_idx < feat_size; feat_idx += blockDim.x) {
        const int idx = base_idx + feat_idx;
        const scalar_t a = anchor[idx];
        const scalar_t p = positive[idx];
        const scalar_t n = negative[idx];
        
        const scalar_t d_pos = a - p;
        const scalar_t d_neg = a - n;
        
        dist_pos += d_pos * d_pos;
        dist_neg += d_neg * d_neg;
    }
    
    // Warp reduction
    #pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        dist_pos += __shfl_down_sync(0xffffffff, dist_pos, offset);
        dist_neg += __shfl_down_sync(0xffffffff, dist_neg, offset);
    }
    
    // First thread in warp writes result
    if (tid == 0) {
        const scalar_t loss = max(scalar_t(0.0), sqrt(dist_pos) - sqrt(dist_neg) + margin);
        output[sample_idx] = loss;
    }
}

torch::Tensor triplet_margin_loss_cuda(
    torch::Tensor anchor,
    torch::Tensor positive,
    torch::Tensor negative,
    float margin) {
    
    TORCH_CHECK(anchor.device().is_cuda(), "anchor must be a CUDA tensor");
    TORCH_CHECK(positive.device().is_cuda(), "positive must be a CUDA tensor");
    TORCH_CHECK(negative.device().is_cuda(), "negative must be a CUDA tensor");
    
    const int batch_size = anchor.size(0);
    const int feat_size = anchor.size(1);
    
    auto output = torch::zeros({batch_size}, anchor.options());
    
    const int threads = 128;
    const int blocks = batch_size;
    
    AT_DISPATCH_FLOATING_TYPES(anchor.scalar_type(), "triplet_margin_loss_kernel", ([&] {
        triplet_margin_loss_kernel<scalar_t><<<blocks, threads>>>(
            anchor.data_ptr<scalar_t>(),
            positive.data_ptr<scalar_t>(),
            negative.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            margin,
            batch_size,
            feat_size);
    }));
    
    return output.mean();
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &triplet_margin_loss_cuda, "Triplet margin loss forward (CUDA)");
}