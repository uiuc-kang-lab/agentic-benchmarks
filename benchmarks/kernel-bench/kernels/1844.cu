#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 128
#define NUM_STREAMS 4

__global__ void triangular_mm_kernel_atomic_minimized(const float* __restrict__ A,
                                              const float* __restrict__ B,
                                              float* __restrict__ C,
                                              int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col <= row) {
        float sum = 0.f;
        for (int k = col; k <= row; ++k) {
            sum += __ldg(&A[row * N + k]) * __ldg(&B[k * N + col]);
        }
        // Reduce use of atomic operations by storing individual results
        // in shared memory and then writing to global memory if thread group size > 1
        if (blockDim.x == 1 && blockDim.y == 1) {
            C[row * N + col] = sum;
        } else {
            atomicAdd(&C[row * N + col], sum);
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::zeros_like(A);  // Initialize with zeros

    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threads - 1) / threads, (N + threads - 1) / threads);

    triangular_mm_kernel_atomic_minimized<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}
