#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// CUDA kernel that computes Smooth L1 (Huber) Loss using vectorized float4 loads and warp-level reductions with __shfl_down_sync.
// Instead of using shared memory for intra-block reductions, each warp performs its own reduction and then its lane 0 atomically adds its value to the global output.

__global__ void smooth_l1_loss_kernel_warp_atomic(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    int n_elements
) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;
    int stride = gridDim.x * blockDim.x;
    float thread_sum = 0.0f;

    // Vectorized processing using float4 loads for 128-bit aligned memory access
    int vec_count = n_elements / 4;  // number of float4 groups
    const float4* pred4 = reinterpret_cast<const float4*>(predictions);
    const float4* targ4 = reinterpret_cast<const float4*>(targets);

    for (int i = idx; i < vec_count; i += stride) {
        float4 p = __ldg(pred4 + i);
        float4 t = __ldg(targ4 + i);
        
        float diff = p.x - t.x;
        float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;

        diff = p.y - t.y;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;

        diff = p.z - t.z;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;

        diff = p.w - t.w;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;
    }

    // Process remaining elements that don't fit into a vectorized load
    int scalar_start = vec_count * 4;
    for (int i = scalar_start + idx; i < n_elements; i += stride) {
        float diff = __ldg(predictions + i) - __ldg(targets + i);
        float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;
    }

    // Warp-level reduction using __shfl_down_sync
    unsigned int mask = 0xffffffff;
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        thread_sum += __shfl_down_sync(mask, thread_sum, offset);
    }

    // Each warp's lane 0 atomically adds its partial sum to the global output.
    if ((tid & (warpSize - 1)) == 0) {
        atomicAdd(output, thread_sum / n_elements);
    }
}

// Host function to launch the kernel
torch::Tensor smooth_l1_loss_cuda_warp_atomic(
    torch::Tensor predictions,
    torch::Tensor targets
) {
    TORCH_CHECK(predictions.sizes() == targets.sizes(), "Input tensors must have the same shape");
    TORCH_CHECK(predictions.is_contiguous() && targets.is_contiguous(), "Input tensors must be contiguous");
    TORCH_CHECK(predictions.device().is_cuda() && targets.device().is_cuda(), "Inputs must be CUDA tensors");

    int n = predictions.numel();
    auto output = torch::zeros({1}, predictions.options());

    const int block_size = 256;
    int vec_count = n / 4;
    int grid_size = (vec_count + block_size - 1) / block_size;
    if (grid_size < 1) grid_size = 1;

    smooth_l1_loss_kernel_warp_atomic<<<grid_size, block_size>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &smooth_l1_loss_cuda_warp_atomic, "Smooth L1 Loss (CUDA) using vectorized loads and warp-level atomic reductions");
}
