#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>

template <typename scalar_t>
__device__ __forceinline__ scalar_t warp_reduce_max(scalar_t val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = max(val, __shfl_down_sync(0xffffffff, val, offset));
    }
    return val;
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t warp_reduce_sum(scalar_t val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

template <typename scalar_t>
__global__ void log_softmax_forward_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int dim_size) {

    const int batch_idx = blockIdx.x;
    const int tid = threadIdx.x;
    const int lane_id = tid % 32;
    const int warp_id = tid / 32;
    const int warps_per_block = blockDim.x / 32;
    
    const scalar_t* __restrict__ input_row = input + batch_idx * dim_size;
    scalar_t* output_row = output + batch_idx * dim_size;

    extern __shared__ __align__(sizeof(scalar_t)) unsigned char smem[];
    scalar_t* warp_max = reinterpret_cast<scalar_t*>(smem);
    scalar_t* warp_sum = warp_max + warps_per_block;

    // Coalesced max value search
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    for (int base = 0; base < dim_size; base += blockDim.x) {
        int idx = base + tid;
        if (idx < dim_size) {
            max_val = max(max_val, __ldg(input_row + idx));
        }
    }

    max_val = warp_reduce_max(max_val);
    if (lane_id == 0) warp_max[warp_id] = max_val;
    __syncthreads();

    if (tid < 32) {
        max_val = (tid < warps_per_block) ? warp_max[tid] : -std::numeric_limits<scalar_t>::infinity();
        max_val = warp_reduce_max(max_val);
        if (tid == 0) warp_max[0] = max_val;
    }
    __syncthreads();
    max_val = warp_max[0];

    // Coalesced exp sum calculation with cached intermediate values
    scalar_t sum = 0;
    scalar_t* temp_exp = reinterpret_cast<scalar_t*>(smem + 2 * warps_per_block * sizeof(scalar_t));
    
    // First pass: compute and cache exponentials
    for (int base = 0; base < dim_size; base += blockDim.x) {
        int idx = base + tid;
        if (idx < dim_size) {
            scalar_t val = __ldg(input_row + idx) - max_val;
            temp_exp[idx] = exp(val);  // Cache the exp calculation
            sum += temp_exp[idx];
        }
    }

    sum = warp_reduce_sum(sum);
    if (lane_id == 0) warp_sum[warp_id] = sum;
    __syncthreads();

    if (tid < 32) {
        sum = (tid < warps_per_block) ? warp_sum[tid] : 0;
        sum = warp_reduce_sum(sum);
        if (tid == 0) warp_sum[0] = sum;
    }
    __syncthreads();

    scalar_t log_sum = log(warp_sum[0]);

    // Coalesced final write using cached exp values
    for (int base = 0; base < dim_size; base += blockDim.x) {
        int idx = base + tid;
        if (idx < dim_size) {
            output_row[idx] = log(temp_exp[idx]) - log_sum;  // Use cached exp value
        }
    }
}

torch::Tensor log_softmax_cuda_forward(torch::Tensor input, int64_t dim) {
    auto ndim = input.dim();
    dim = dim >= 0 ? dim : dim + ndim;

    std::vector<int64_t> permute_dims;
    for (int64_t i = 0; i < ndim; ++i) {
        if (i != dim) permute_dims.push_back(i);
    }
    permute_dims.push_back(dim);

    input = input.permute(permute_dims).contiguous();
    auto output = torch::empty_like(input);
    
    int64_t batch_size = input.numel() / input.size(-1);
    int64_t dim_size = input.size(-1);

    int threads = 256;
    while (threads > dim_size && threads > 32) threads /= 2;
    threads = ((threads + 31) / 32) * 32;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "log_softmax_forward_cuda", ([&] {
        size_t shared_mem_size = (threads/32) * sizeof(scalar_t) * 2;
        log_softmax_forward_kernel<scalar_t><<<batch_size, threads, shared_mem_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            dim_size);
    }));

    std::vector<int64_t> inverse_permute_dims(ndim);
    for (size_t i = 0; i < permute_dims.size(); ++i) {
        inverse_permute_dims[permute_dims[i]] = i;
    }
    return output.permute(inverse_permute_dims);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &log_softmax_cuda_forward, "LogSoftmax forward (CUDA)");
}