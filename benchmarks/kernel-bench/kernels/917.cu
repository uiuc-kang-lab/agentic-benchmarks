#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor matmul_cuda(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0;
    const float beta = 0.0;
    
    // Proper row-major handling through transposed column-major input
    hipblasSgemm(handle,
                HIPBLAS_OP_T,   // A is treated as KxM (original is row-major MxK)
                HIPBLAS_OP_T,   // B is treated as NxK (original is row-major KxN)
                M, N, K,
                &alpha,
                A.data_ptr<float>(), K,
                B.data_ptr<float>(), N,
                &beta,
                C.data_ptr<float>(), M);

    hipblasDestroy(handle);
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_cuda, "cuBLAS Matrix Multiplication (CUDA)");
}