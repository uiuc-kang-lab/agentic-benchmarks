#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ __forceinline__ float4 load_float4(const float* ptr) {
    return *reinterpret_cast<const float4*>(ptr);
}

__device__ __forceinline__ void store_float4(float* ptr, float4 val) {
    *reinterpret_cast<float4*>(ptr) = val;
}

__global__ void depthwise_conv2d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int in_h,
    int in_w,
    int out_channels,
    int out_h,
    int out_w,
    int kernel_h,
    int kernel_w,
    int stride_h,
    int stride_w,
    int padding_h,
    int padding_w,
    int dilation_h,
    int dilation_w,
    int groups,
    int channels_per_group
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int elements_per_thread = 4;
    const int total_elements = batch_size * out_channels * out_h * out_w;
    
    for (int i = tid * elements_per_thread; i < total_elements; i += blockDim.x * gridDim.x * elements_per_thread) {
        int remaining = i;
        const int w_out_base = remaining % out_w;
        remaining /= out_w;
        const int h_out = remaining % out_h;
        remaining /= out_h;
        const int c_out = remaining % out_channels;
        const int b = remaining / out_channels;

        if (b >= batch_size) continue;

        const int g = c_out / channels_per_group;
        const int m = c_out % channels_per_group;

        const int input_batch_offset = b * in_channels * in_h * in_w;
        const int input_channel_offset = g * in_h * in_w;
        const int weight_offset = (g * channels_per_group + m) * kernel_h * kernel_w;

        #pragma unroll
        for (int elem = 0; elem < elements_per_thread && w_out_base + elem < out_w; ++elem) {
            const int w_out = w_out_base + elem;
            float sum = 0.0f;

            #pragma unroll
            for (int kh = 0; kh < kernel_h; ++kh) {
                const int h_in = h_out * stride_h - padding_h + kh * dilation_h;
                if (h_in >= 0 && h_in < in_h) {
                    #pragma unroll
                    for (int kw = 0; kw < kernel_w; ++kw) {
                        const int w_in = w_out * stride_w - padding_w + kw * dilation_w;
                        if (w_in >= 0 && w_in < in_w) {
                            const int input_idx = input_batch_offset + input_channel_offset + h_in * in_w + w_in;
                            const int weight_idx = weight_offset + kh * kernel_w + kw;
                            sum += input[input_idx] * weight[weight_idx];
                        }
                    }
                }
            }

            if (bias != nullptr) {
                sum += bias[c_out];
            }

            const int out_idx = ((b * out_channels + c_out) * out_h + h_out) * out_w + w_out;
            output[out_idx] = sum;
        }
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    int stride_h,
    int stride_w,
    int padding_h,
    int padding_w,
    int dilation_h,
    int dilation_w,
    int groups
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int in_h = x.size(2);
    int in_w = x.size(3);

    int kernel_h = weight.size(2);
    int kernel_w = weight.size(3);
    int out_channels = groups * weight.size(1);
    int channels_per_group = out_channels / groups;

    int out_h = (in_h + 2 * padding_h - dilation_h * (kernel_h - 1) - 1) / stride_h + 1;
    int out_w = (in_w + 2 * padding_w - dilation_w * (kernel_w - 1) - 1) / stride_w + 1;

    auto output = torch::empty({batch_size, out_channels, out_h, out_w}, x.options());

    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        bias_ptr = bias->data_ptr<float>();
    }

    const int elements_per_thread = 4;
    const int total_elements = batch_size * out_channels * out_h * out_w;
    const int threads = 256;
    const int blocks = (total_elements + (threads * elements_per_thread) - 1) / (threads * elements_per_thread);

    depthwise_conv2d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        in_h,
        in_w,
        out_channels,
        out_h,
        out_w,
        kernel_h,
        kernel_w,
        stride_h,
        stride_w,
        padding_h,
        padding_w,
        dilation_h,
        dilation_w,
        groups,
        channels_per_group
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Depthwise Conv2D forward (CUDA)");
}