#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void elu_kernel_optimized(const float* x, float* out, float alpha, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float input_val = x[idx];
        float result = (input_val > 0) ? input_val : alpha * (expf(input_val) - 1);
        out[idx] = result;
        // atomic operations are not needed in this context as there are no race conditions.
    }
}

torch::Tensor elu_cuda_optimized(torch::Tensor x, float alpha) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int n = x.numel();

    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;

    elu_kernel_optimized<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(x.data_ptr<float>(), out.data_ptr<float>(), alpha, n);

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &elu_cuda_optimized, "Optimized ELU activation (CUDA)");
}