#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Compute GELU for a single scalar value
__device__ __forceinline__ float gelu_scalar(float xi) {
    const float sqrt_2_over_pi = 0.7978845608f;
    const float coeff = 0.044715f;
    float x2 = xi * xi;
    float x3 = x2 * xi;
    float inner = sqrt_2_over_pi * (xi + coeff * x3);
    return 0.5f * xi * (1.f + tanhf(inner));
}

// Kernel that evenly distributes work among threads using balanced chunking
__global__ void gelu_kernel(const float* x, float* y, const int n) {
    extern __shared__ float shared_mem[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int totalThreads = gridDim.x * blockDim.x;

    // Process vectorized portion using float4 loads and stores
    int totalVectors = n / 4;  // number of complete groups of 4
    int vecChunk = (totalVectors + totalThreads - 1) / totalThreads;
    int startVec = tid * vecChunk;
    int endVec = startVec + vecChunk;
    if (endVec > totalVectors) endVec = totalVectors;

    for (int i = startVec; i < endVec; i++) {
        const float4* x_vec = reinterpret_cast<const float4*>(x);
        float4* y_vec = reinterpret_cast<float4*>(y);
        float4 in = x_vec[i];
        float4 out;
        out.x = gelu_scalar(in.x);
        out.y = gelu_scalar(in.y);
        out.z = gelu_scalar(in.z);
        out.w = gelu_scalar(in.w);
        y_vec[i] = out;
    }

    // Handle remaining elements that don't form a complete float4
    int remainder_start = totalVectors * 4;
    int totalRemainder = n - remainder_start;
    int remChunk = (totalRemainder + totalThreads - 1) / totalThreads;
    int startRem = tid * remChunk;
    int endRem = startRem + remChunk;
    if (endRem > totalRemainder) endRem = totalRemainder;

    for (int i = startRem; i < endRem; i++) {
        int idx = remainder_start + i;
        y[idx] = gelu_scalar(x[idx]);
    }
}

torch::Tensor gelu_forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(x.is_contiguous(), "Input tensor must be contiguous");

    auto y = torch::empty_like(x);
    int n = x.numel();

    const int threads = 256;
    int numSMs = 0;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
    int max_blocks = numSMs * 32;
    int blocks = (n + threads - 1) / threads;
    if (blocks > max_blocks) {
        blocks = max_blocks;
    }

    gelu_kernel<<<blocks, threads>>>(x.data_ptr<float>(), y.data_ptr<float>(), n);
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &gelu_forward, "GELU forward CUDA implementation");
}
