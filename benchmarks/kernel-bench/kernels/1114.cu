#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define TILE_SIZE 32

template <typename scalar_t>
__global__ void module_fn_cuda_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ output,
    int N, int M, int K, int L) {

    __shared__ scalar_t B_shared[TILE_SIZE][TILE_SIZE];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int n = by;
    int m = tx;
    
    if (n < N && m < M) {
        for (int l = 0; l < L; l++) {
            scalar_t sum = 0;
            
            for (int k_tile = 0; k_tile < K; k_tile += TILE_SIZE) {
                // Collaborative loading of B into shared memory
                if (ty + k_tile < K && tx < L)
                    B_shared[ty][tx] = B[(ty + k_tile) * L + tx];
                __syncthreads();
                
                // Compute partial sum
                #pragma unroll
                for (int k = 0; k < TILE_SIZE && k + k_tile < K; k++) {
                    sum += A[n * M * K + m * K + (k + k_tile)] * B_shared[k][0];
                }
                __syncthreads();
            }
            
            output[n * M * L + m * L + l] = sum;
        }
    }
}

void module_fn_cuda_forward(
    torch::Tensor A,
    torch::Tensor B,
    torch::Tensor output) {

    int N = A.size(0);
    int M = A.size(1);
    int K = A.size(2);
    int L = B.size(1);

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((M + threads.x - 1) / threads.x, 
                (N + threads.y - 1) / threads.y);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(A.scalar_type(), "module_fn_cuda_forward", ([&] {
        module_fn_cuda_kernel<scalar_t><<<blocks, threads>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            N, M, K, L);
    }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in module_fn_cuda_forward: %s\n", hipGetErrorString(err));
    }
}

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor module_fn_forward(
    torch::Tensor A,
    torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    auto N = A.size(0);
    auto M = A.size(1);
    auto L = B.size(1);

    auto output = torch::zeros({N, M, L}, A.options());
    module_fn_cuda_forward(A, B, output);
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &module_fn_forward, "module_fn forward (CUDA)");
}