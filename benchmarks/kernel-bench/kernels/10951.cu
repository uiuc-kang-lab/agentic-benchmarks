#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static const int BLOCK_SIZE = 256;

template <typename scalar_t>
__global__ void mse_forward_kernel_stride(
    const scalar_t* __restrict__ preds,
    const scalar_t* __restrict__ tgts,
    double* __restrict__ sum_out,
    const int64_t num_elements
) {
    __shared__ double shm[BLOCK_SIZE];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double thread_sum = 0.0;

    // Strided loop to accumulate squared differences
    for (int i = idx; i < num_elements; i += blockDim.x * gridDim.x) {
        double diff = static_cast<double>(preds[i]) - static_cast<double>(tgts[i]);
        thread_sum += diff * diff;
    }

    // Store partial sums in shared memory
    shm[threadIdx.x] = thread_sum;
    __syncthreads();

    // Block-level reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shm[threadIdx.x] += shm[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Add the reduced sum from this block into global accumulator
    if (threadIdx.x == 0) {
        atomicAdd(sum_out, shm[0]);
    }
}

torch::Tensor forward_stride(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.is_cuda(), "predictions must be a CUDA tensor");
    TORCH_CHECK(targets.is_cuda(), "targets must be a CUDA tensor");
    TORCH_CHECK(predictions.numel() == targets.numel(),
                "predictions and targets must have the same number of elements");

    const int64_t num_elements = predictions.numel();
    // Use double for accumulation
    auto accumulator = torch::zeros({1}, predictions.options().dtype(at::kDouble));

    const int grid_size = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    AT_DISPATCH_FLOATING_TYPES(predictions.scalar_type(), "mse_forward_stride_cuda", [&] {
        mse_forward_kernel_stride<scalar_t><<<grid_size, BLOCK_SIZE>>>(
            predictions.data_ptr<scalar_t>(),
            targets.data_ptr<scalar_t>(),
            accumulator.data_ptr<double>(),
            num_elements
        );
    });

    // Final mean = accumulator / N
    auto result = accumulator.div_(static_cast<double>(num_elements));
    return result.to(predictions.dtype());
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward_stride", &forward_stride, "Mean Squared Error (MSE) forward with stride (CUDA)");
}