#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

__global__ void upper_triangular_matmul_kernel(const float* __restrict__ A,
                                                 const float* __restrict__ B,
                                                 float* __restrict__ C,
                                                 int N) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int lane_id = threadIdx.x;
    const int col = row + lane_id;

    if (row >= N || col >= N || row > col) return;

    float sum = 0.0f;
    
    // Each thread computes its own summation range from row to col
    for (int k = row; k <= col; ++k) {
        // Let thread with lane_id 0 load A[row,k] and broadcast it
        float a_val = (lane_id == 0) ? A[row * N + k] : 0.0f;
        a_val = __shfl_sync(0xffffffff, a_val, 0);

        float b_val = __ldg(&B[k * N + col]);
        sum += a_val * b_val;
    }

    C[row * N + col] = sum;
}

torch::Tensor upper_triangular_matmul(torch::Tensor A, torch::Tensor B) {
    int N = A.size(0);
    auto C = torch::zeros_like(A);

    // Each block processes 8 rows with 32 threads per row
    dim3 threads(32, 8);
    dim3 blocks(1, (N + threads.y - 1) / threads.y);

    upper_triangular_matmul_kernel<<<blocks, threads>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &upper_triangular_matmul, "Warp-optimized upper triangular matmul");
}