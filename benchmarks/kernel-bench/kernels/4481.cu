#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/cuda/HIPContext.h>
#include <vector>

// Compute Statistics Kernel with Stride Loop for handling large workloads
// Each block computes the mean and variance for one (sample, group) pair

template <typename scalar_t>
__global__ void compute_stats_kernel(
    const scalar_t* __restrict__ x,
    const int N,
    const int C,
    const int spatial,             // product of dimensions from index 2 onward
    const int channels_per_group,  // C / num_groups
    const int num_groups,
    scalar_t* __restrict__ mean,   // output shape: (N, num_groups)
    scalar_t* __restrict__ var) {  // output shape: (N, num_groups)

  // Decode block index: each block is responsible for one (n, g) pair
  int idx = blockIdx.x;
  int n = idx / num_groups;
  int g = idx % num_groups;

  int group_offset = n * C * spatial + g * channels_per_group * spatial;
  int group_elems = channels_per_group * spatial;

  scalar_t sum = 0;
  scalar_t sum_sq = 0;

  // Use stride loop to cover all elements in this group
  for (int i = threadIdx.x; i < group_elems; i += blockDim.x) {
    int c = i / spatial;
    int j = i % spatial;
    int index = group_offset + c * spatial + j;
    scalar_t val = x[index];
    sum += val;
    sum_sq += val * val;
  }

  // Reduction using shared memory
  extern __shared__ char smem[];
  scalar_t* s_sum = reinterpret_cast<scalar_t*>(smem);
  scalar_t* s_sum_sq = s_sum + blockDim.x;

  s_sum[threadIdx.x] = sum;
  s_sum_sq[threadIdx.x] = sum_sq;
  __syncthreads();

  // Standard reduction loop
  for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
    if (threadIdx.x < stride) {
      s_sum[threadIdx.x] += s_sum[threadIdx.x + stride];
      s_sum_sq[threadIdx.x] += s_sum_sq[threadIdx.x + stride];
    }
    __syncthreads();
  }

  // First thread writes the result
  if (threadIdx.x == 0) {
    scalar_t group_mean = s_sum[0] / group_elems;
    scalar_t group_var = s_sum_sq[0] / group_elems - group_mean * group_mean;
    int out_index = n * num_groups + g;
    mean[out_index] = group_mean;
    var[out_index] = group_var;
  }
}

// Group Norm Forward Kernel with Grid-Stride Loop for proper boundary handling

template <typename scalar_t>
__global__ void group_norm_forward_kernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ mean,
    const scalar_t* __restrict__ var,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    const int N,
    const int C,
    const int spatial,             // product of dimensions from index 2 onward
    const int channels_per_group,  // C / num_groups
    const int num_groups,
    const scalar_t eps,
    scalar_t* __restrict__ y) {

  int total = N * C * spatial;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  // Grid-stride loop to cover all elements
  for (int idx = tid; idx < total; idx += stride) {
    int j = idx % spatial;
    int temp = idx / spatial;
    int c = temp % C;
    int n = temp / C;
    int g = c / channels_per_group;

    int stats_index = n * num_groups + g;
    scalar_t m = mean[stats_index];
    scalar_t v = var[stats_index];
    scalar_t inv_std = rsqrt(v + eps);
    scalar_t x_val = x[idx];
    y[idx] = (x_val - m) * inv_std * weight[c] + bias[c];
  }
}

// Host function to launch both kernels

torch::Tensor group_norm_forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    int64_t num_groups,
    double eps) {

  const int N = x.size(0);
  const int C = x.size(1);
  int spatial = 1;
  for (int d = 2; d < x.dim(); d++) {
    spatial *= x.size(d);
  }
  const int channels_per_group = C / num_groups;

  auto y = torch::empty_like(x);
  auto options = torch::TensorOptions().device(x.device()).dtype(x.dtype());
  auto mean = torch::empty({N, num_groups}, options);
  auto var = torch::empty({N, num_groups}, options);

  // Launch compute_stats_kernel: one block per group per sample (total_groups = N * num_groups)
  int total_groups = N * num_groups;
  int threads_stats = 256;
  dim3 blocks_stats(total_groups);
  size_t shared_mem_size = threads_stats * 2 * sizeof(float);

  // Launch group_norm_forward_kernel with grid-stride loop
  int total_elements = N * C * spatial;
  int threads_norm = 256;
  int blocks_norm = (total_elements + threads_norm - 1) / threads_norm;

  hipStream_t stream = c10::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "group_norm_forward_cuda", ([&] {
    compute_stats_kernel<scalar_t><<<blocks_stats, threads_stats, shared_mem_size, stream>>>(
        x.data_ptr<scalar_t>(),
        N,
        C,
        spatial,
        channels_per_group,
        num_groups,
        mean.data_ptr<scalar_t>(),
        var.data_ptr<scalar_t>());

    group_norm_forward_kernel<scalar_t><<<blocks_norm, threads_norm, 0, stream>>>(
        x.data_ptr<scalar_t>(),
        mean.data_ptr<scalar_t>(),
        var.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(),
        bias.data_ptr<scalar_t>(),
        N,
        C,
        spatial,
        channels_per_group,
        num_groups,
        static_cast<scalar_t>(eps),
        y.data_ptr<scalar_t>());
  }));

  return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &group_norm_forward, "Group Normalization forward (CUDA) with stride loops");
}
