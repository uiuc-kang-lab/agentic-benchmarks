#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            float sum = 0.f;
            // Use aligned memory access pattern with __ldg for read-only
            #pragma unroll(4)
            for (int k = col; k <= row; ++k) {
                sum += __ldg(&A[row * N + k]) * __ldg(&B[k * N + col]);
            }
            // 128-bit aligned store
            *reinterpret_cast<float4*>(&C[row * N + col]) = make_float4(sum, 0.f, 0.f, 0.f);
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(A.size(0) == A.size(1) && B.size(0) == B.size(1), "Matrices must be square");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    const int threads = 32;
    dim3 blocks((N + threads-1)/threads, (N + threads-1)/threads);
    triangular_mm_kernel<<<blocks, dim3(threads, threads)>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA error: ", hipGetErrorString(err));
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized triangular matmul with LDG");
}