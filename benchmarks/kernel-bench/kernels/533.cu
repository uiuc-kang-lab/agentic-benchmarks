#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

// This kernel combines vectorized memory accesses via float4 with loop unrolling in a grid-stride loop
// to improve memory throughput and reduce loop overhead.

__global__ void multiplyKernelCombined(const float* __restrict__ A,
                                         float* __restrict__ C,
                                         float s,
                                         int64_t size) {
    // Total number of threads
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process most of the data in groups of 4 floats using vectorized memory accesses
    int vectorCount = size / 4;  // Number of groups of 4 elements

    // Use grid-stride loop and unroll 4 iterations per loop to reduce loop overhead
    for (int i = tid; i < vectorCount; i += stride * 4) {
        // If the next 3 vector loads are within bounds, unroll without per-iteration bounds checking
        if (i + 3 * stride < vectorCount) {
            float4 a0 = __ldg(reinterpret_cast<const float4*>(A) + i);
            float4 a1 = __ldg(reinterpret_cast<const float4*>(A) + i + stride);
            float4 a2 = __ldg(reinterpret_cast<const float4*>(A) + i + 2 * stride);
            float4 a3 = __ldg(reinterpret_cast<const float4*>(A) + i + 3 * stride);
            a0.x *= s; a0.y *= s; a0.z *= s; a0.w *= s;
            a1.x *= s; a1.y *= s; a1.z *= s; a1.w *= s;
            a2.x *= s; a2.y *= s; a2.z *= s; a2.w *= s;
            a3.x *= s; a3.y *= s; a3.z *= s; a3.w *= s;
            reinterpret_cast<float4*>(C)[i] = a0;
            reinterpret_cast<float4*>(C)[i + stride] = a1;
            reinterpret_cast<float4*>(C)[i + 2 * stride] = a2;
            reinterpret_cast<float4*>(C)[i + 3 * stride] = a3;
        } else {
            #pragma unroll
            for (int offset = 0; offset < 4; offset++) {
                int idx = i + offset * stride;
                if (idx < vectorCount) {
                    float4 a = __ldg(reinterpret_cast<const float4*>(A) + idx);
                    a.x *= s;
                    a.y *= s;
                    a.z *= s;
                    a.w *= s;
                    reinterpret_cast<float4*>(C)[idx] = a;
                }
            }
        }
    }

    // Process the tail elements that are not a multiple of 4
    int tailStart = vectorCount * 4;
    for (int i = tailStart + tid; i < size; i += stride) {
        C[i] = A[i] * s;
    }
}

// Torch binding: wraps the kernel in a forward function

torch::Tensor forward(torch::Tensor A, float s) {
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor.");
    TORCH_CHECK(A.scalar_type() == torch::kFloat, "Input tensor A must be of type float.");

    auto C = torch::empty_like(A);
    int64_t size = A.numel();

    // Compute the number of vectorized (float4) elements
    int vectorCount = size / 4;
    const int threads = 256;
    // Use the vectorized portion to determine grid configuration
    int blocks = (vectorCount + threads - 1) / threads;

    multiplyKernelCombined<<<blocks, threads>>>(A.data_ptr<float>(),
                                                  C.data_ptr<float>(),
                                                  s,
                                                  size);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Efficient combined matrix-scalar multiplication kernel");
}
