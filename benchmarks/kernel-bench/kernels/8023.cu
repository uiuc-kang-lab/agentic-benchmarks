#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK((x).is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK((x).is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Declare constant memory for weights (assuming max size of 1024 floats)
__constant__ float d_weights[1024];

// CUDA kernel using stride loops to handle workloads larger than available threads
__global__ void stride_loop_conv1d_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    const float* bias,
    int batch_size,
    int in_channels,
    int out_channels,
    int input_width,
    int output_width,
    int kernel_size,
    int stride,
    int padding,
    int groups,
    int total_elements) {

    // Compute global thread index and stride
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride_val = blockDim.x * gridDim.x;

    // Each thread processes multiple elements using a stride loop
    for (int idx = tid; idx < total_elements; idx += stride_val) {
        // Calculate output indices
        int j = idx % output_width;                       // output spatial position
        int o = (idx / output_width) % out_channels;        // output channel
        int b = idx / (output_width * out_channels);        // batch index

        float sum = 0.0f;
        int group_in_channels = in_channels / groups;
        int group_size_out = out_channels / groups;
        int g = o / group_size_out;  // group index

        // Iterate over kernel elements
        for (int k = 0; k < kernel_size; k++) {
            int i = j + padding - k;
            if (i % stride != 0) continue;
            i /= stride;
            if (i < 0 || i >= input_width) continue;

            // Accumulate over relevant input channels
            for (int ic = 0; ic < group_in_channels; ic++) {
                int real_ic = g * group_in_channels + ic;
                int input_idx = b * in_channels * input_width + real_ic * input_width + i;
                int weight_idx = (real_ic * group_size_out + (o - g * group_size_out)) * kernel_size + k;
                sum += input[input_idx] * d_weights[weight_idx];
            }
        }

        // Add bias if provided
        if (bias != nullptr) {
            sum += bias[o];
        }
        
        output[idx] = sum;
    }
}

// Host wrapper function
torch::Tensor forward(
    const torch::Tensor& x,
    const torch::Tensor& weight,
    const c10::optional<torch::Tensor>& bias,
    int64_t stride,
    int64_t padding,
    int64_t output_padding,
    int64_t groups) {

    CHECK_INPUT(x);
    CHECK_INPUT(weight);

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int input_width = x.size(2);
    int kernel_size = weight.size(2);
    int group_size_out = weight.size(1);
    int out_channels = group_size_out * groups;

    // Compute output width based on transposed convolution formula
    int output_width = (input_width - 1) * stride - 2 * padding + kernel_size + output_padding;

    auto output = torch::zeros({batch_size, out_channels, output_width}, x.options());

    // Copy weight tensor to constant memory for fast read-only access
    int num_weight_elems = weight.numel();
    TORCH_CHECK(num_weight_elems <= 1024, "Weight size exceeds constant memory limit");
    hipMemcpyToSymbol(HIP_SYMBOL(d_weights), weight.data_ptr<float>(), num_weight_elems * sizeof(float), 0, hipMemcpyDeviceToDevice);

    int total_elements = batch_size * out_channels * output_width;
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;

    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
        bias_ptr = bias.value().data_ptr<float>();
    }

    stride_loop_conv1d_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        x.data_ptr<float>(),
        output.data_ptr<float>(),
        bias_ptr,
        batch_size,
        in_channels,
        out_channels,
        input_width,
        output_width,
        kernel_size,
        stride,
        padding,
        groups,
        total_elements
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Transposed 1D convolution forward (CUDA) using stride loops for extended workloads");
}
