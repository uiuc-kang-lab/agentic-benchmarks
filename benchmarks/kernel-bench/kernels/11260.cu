#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void smooth_l1_loss_kernel(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    int n_elements
) {
    const int elements_per_thread = 8;  // Increased from 4 to 8
    const int tid = threadIdx.x;
    const int block_offset = blockIdx.x * blockDim.x * elements_per_thread;
    const int idx_start = block_offset + tid * elements_per_thread;
    float thread_sum = 0.0f;

    #pragma unroll
    for (int j = 0; j < elements_per_thread; j++) {
        int idx = idx_start + j;
        if (idx < n_elements) {
            float diff = predictions[idx] - targets[idx];
            float abs_diff = fabsf(diff);
            thread_sum += (abs_diff < 1.0f) ? (0.5f * diff * diff) : (abs_diff - 0.5f);
        }
    }

    __shared__ float shared_sum[128];  // Reduced shared memory size
    shared_sum[tid] = thread_sum;
    __syncthreads();

    // Optimized reduction for 128 threads
    if (tid < 64) shared_sum[tid] += shared_sum[tid + 64];
    __syncthreads();
    
    if (tid < 32) {
        volatile float* smem = shared_sum;
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }

    if (tid == 0) {
        atomicAdd(output, shared_sum[0] / n_elements);
    }
}

torch::Tensor smooth_l1_loss_cuda(
    torch::Tensor predictions,
    torch::Tensor targets
) {
    TORCH_CHECK(
        predictions.sizes() == targets.sizes(),
        "Input tensors must have the same shape"
    );
    TORCH_CHECK(
        predictions.is_contiguous() && targets.is_contiguous(),
        "Input tensors must be contiguous"
    );
    TORCH_CHECK(
        predictions.device().is_cuda() && targets.device().is_cuda(),
        "Inputs must be CUDA tensors"
    );

    int n = predictions.numel();
    auto output = torch::zeros({1}, predictions.options());

    const int block_size = 128;  // Optimized block size
    const int elements_per_thread = 8;
    const int grid_size = (n + block_size * elements_per_thread - 1) / (block_size * elements_per_thread);

    smooth_l1_loss_kernel<<<grid_size, block_size>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &smooth_l1_loss_cuda, "Smooth L1 Loss (CUDA)");
}