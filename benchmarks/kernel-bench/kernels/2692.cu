#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Kernel without shared memory and without unnecessary __syncthreads()
// Each thread independently processes elements via a grid-stride loop, relying on coalesced global accesses.
__global__ void leaky_relu_nosync_kernel(const float* __restrict__ x, float* __restrict__ out, float negative_slope, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = idx; i < n; i += stride) {
        float val = x[i];
        out[i] = val > 0.0f ? val : val * negative_slope;
    }
}

// Forward function that launches the kernel
torch::Tensor leaky_relu_forward(torch::Tensor x, float negative_slope) {
    CHECK_INPUT(x);
    auto out = torch::empty_like(x);
    int n = x.numel();

    const int threads = 1024;
    const int blocks = (n + threads - 1) / threads;

    leaky_relu_nosync_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(), out.data_ptr<float>(), negative_slope, n
    );

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &leaky_relu_forward, "LeakyReLU forward without unnecessary synchronizations (CUDA)");
}
