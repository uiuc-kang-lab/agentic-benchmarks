#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel that uses warp-level primitives with loop unrolling and adaptive block size selection.
// Each warp computes one output element by summing across the reduction dimension.

template <typename scalar_t>
__global__ void adaptive_warp_reduce_sum_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t reduce_size,
    int64_t inner_size,
    int64_t total_outputs) {

    const int warpSize = 32;
    int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = global_thread_id / warpSize;
    int lane = global_thread_id % warpSize;

    // Total number of warps across the grid
    int total_warps = (gridDim.x * blockDim.x) / warpSize;

    // Each warp processes output elements in a grid-stride loop
    for (int out_idx = warp_id; out_idx < total_outputs; out_idx += total_warps) {
        // Map the 1D output index to outer and inner indices
        int outer_idx = out_idx / inner_size;
        int inner_idx = out_idx % inner_size;

        // Compute base index for current reduction
        int64_t base = outer_idx * reduce_size * inner_size + inner_idx;
        scalar_t sum_val = 0;

        // Use loop unrolling to reduce loop overhead
        #pragma unroll
        for (int i = lane; i < reduce_size; i += warpSize) {
            sum_val += input[base + i * inner_size];
        }

        // Warp-level reduction using shuffle down
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            sum_val += __shfl_down_sync(0xFFFFFFFF, sum_val, offset);
        }

        // The first lane writes the result
        if (lane == 0) {
            output[out_idx] = sum_val;
        }
    }
}

// CUDA wrapper function
torch::Tensor sum_reduce_cuda(torch::Tensor input, int64_t dim) {
    // Adjust for negative dimensions
    if (dim < 0) {
        dim += input.dim();
    }

    // Calculate reduction sizes
    auto sizes = input.sizes().vec();
    int64_t reduce_size = sizes[dim];
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }
    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }

    // Prepare output tensor (set reduction dimension to 1)
    sizes[dim] = 1;
    auto output = torch::empty(sizes, input.options());

    // Total number of output elements
    int64_t total_outputs = outer_size * inner_size;

    // Each output element is computed by one warp (32 threads)
    const int warpSize = 32;
    int total_threads = total_outputs * warpSize;

    // Adaptive block size selection from candidates {512, 256, 128, 64, 32}
    int candidate_sizes[5] = {512, 256, 128, 64, 32};
    int block_size = 32; // default
    for (int i = 0; i < 5; i++) {
        if (total_threads >= candidate_sizes[i]) {
            block_size = candidate_sizes[i];
            break;
        }
    }

    int blocks = (total_threads + block_size - 1) / block_size;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sum_reduce_cuda", ([&] {
        adaptive_warp_reduce_sum_kernel<scalar_t><<<blocks, block_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            reduce_size,
            inner_size,
            total_outputs
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &sum_reduce_cuda, "Sum reduction forward (CUDA) with adaptive block size");
}
