#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>
#include <cmath>

// Tunable block size kernel: one block per output element, using shared memory reduction
// Each block processes one output pooling window. The number of threads per block (block size) can be tuned (32, 64, 128, 256, 512) based on hardware.

template <typename scalar_t>
__global__ void max_pool3d_tunable_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t* __restrict__ indices,
    const int batch_size,
    const int channels,
    const int input_d, const int input_h, const int input_w,
    const int output_d, const int output_h, const int output_w,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation) {

    // Each block corresponds to one output element
    const int out_idx = blockIdx.x;
    if (out_idx >= batch_size * channels * output_d * output_h * output_w) return;

    // Decode the linear output index into 5D coordinates: b, c, d_out, h_out, w_out
    const int w_out = out_idx % output_w;
    const int h_out = (out_idx / output_w) % output_h;
    const int d_out = (out_idx / (output_w * output_h)) % output_d;
    const int c = (out_idx / (output_w * output_h * output_d)) % channels;
    const int b = out_idx / (output_w * output_h * output_d * channels);

    // Compute the starting indices in the input volume
    const int d_start = d_out * stride - padding;
    const int h_start = h_out * stride - padding;
    const int w_start = w_out * stride - padding;

    // Total number of elements in the pooling window
    const int poolingVolume = kernel_size * kernel_size * kernel_size;

    // Each thread computes a local maximum over its assigned pooling indices
    scalar_t local_max = -std::numeric_limits<scalar_t>::infinity();
    int64_t local_index = -1;

    for (int i = threadIdx.x; i < poolingVolume; i += blockDim.x) {
        int k_d = i / (kernel_size * kernel_size);
        int rem = i % (kernel_size * kernel_size);
        int k_h = rem / kernel_size;
        int k_w = rem % kernel_size;

        int d_in = d_start + k_d * dilation;
        int h_in = h_start + k_h * dilation;
        int w_in = w_start + k_w * dilation;

        if (d_in >= 0 && d_in < input_d &&
            h_in >= 0 && h_in < input_h &&
            w_in >= 0 && w_in < input_w) {
            int input_idx = ((b * channels + c) * input_d + d_in) * input_h * input_w +
                            h_in * input_w + w_in;
            scalar_t val = input[input_idx];
            if (val > local_max) {
                local_max = val;
                local_index = input_idx;
            }
        }
    }

    // Allocate shared memory for block-level reduction
    // Shared memory layout: first blockDim.x elements for values, next blockDim.x for indices
    extern __shared__ char shared_mem[];
    scalar_t* sdata_val = reinterpret_cast<scalar_t*>(shared_mem);
    int64_t* sdata_idx = reinterpret_cast<int64_t*>(&sdata_val[blockDim.x]);

    sdata_val[threadIdx.x] = local_max;
    sdata_idx[threadIdx.x] = local_index;
    __syncthreads();

    // Perform reduction in two stages: first a shared memory reduction, then a warp-level reduction using shuffle intrinsics for improved performance
    // Stage 1: Reduce to 32 elements using shared memory
    for (int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (threadIdx.x < s) {
            if (sdata_val[threadIdx.x + s] > sdata_val[threadIdx.x]) {
                sdata_val[threadIdx.x] = sdata_val[threadIdx.x + s];
                sdata_idx[threadIdx.x] = sdata_idx[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    // Stage 2: Warp-level reduction using shuffle intrinsics
    if (blockDim.x > 32 && threadIdx.x < 32) {
        // Use shuffle to reduce within the warp; no __syncthreads() needed in warp-synchronous execution
        for (int offset = 16; offset > 0; offset /= 2) {
            scalar_t other_val = __shfl_down_sync(0xffffffff, sdata_val[threadIdx.x], offset);
            int64_t other_idx = __shfl_down_sync(0xffffffff, sdata_idx[threadIdx.x], offset);
            if (other_val > sdata_val[threadIdx.x]) {
                sdata_val[threadIdx.x] = other_val;
                sdata_idx[threadIdx.x] = other_idx;
            }
        }
    }

    // Write the result for this output element (only thread 0 writes out the final maximum)
    if (threadIdx.x == 0) {
        output[out_idx] = sdata_val[0];
        if (indices != nullptr) {
            indices[out_idx] = sdata_idx[0];
        }
    }
}

// Host wrapper function for the tunable max pooling kernel

torch::Tensor max_pool3d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation,
    bool return_indices,
    bool ceil_mode) {

    auto input_sizes = input.sizes();
    const int batch_size = input_sizes[0];
    const int channels = input_sizes[1];
    const int input_d = input_sizes[2];
    const int input_h = input_sizes[3];
    const int input_w = input_sizes[4];

    // Compute output dimensions
    const int output_d = ceil_mode ?
        static_cast<int>(ceil((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1)) :
        static_cast<int>(floor((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1));
    const int output_h = ceil_mode ?
        static_cast<int>(ceil((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1)) :
        static_cast<int>(floor((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1));
    const int output_w = ceil_mode ?
        static_cast<int>(ceil((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1)) :
        static_cast<int>(floor((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1));

    auto output = torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options());
    auto indices = return_indices ?
        torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options().dtype(torch::kLong)) :
        torch::Tensor();

    // Total number of output elements
    const int total_outputs = batch_size * channels * output_d * output_h * output_w;

    // Experiment with block sizes. Try one of: 32, 64, 128, 256, 512.
    // Tunable parameter. For many cases, smaller pooling windows favor lower thread counts.
    const int threads = 64; // Change this value to experiment with different block sizes
    const int blocks = total_outputs; // One block per output element
    
    // Compute shared memory size: each block allocates 'threads' elements for values and the same for indices
    size_t shared_mem_size = threads * (sizeof(float) + sizeof(int64_t));
    // Note: if input is double, adjust accordingly using input.element_size() later
    if (input.scalar_type() == torch::kDouble) {
        shared_mem_size = threads * (sizeof(double) + sizeof(int64_t));
    }

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool3d_forward_cuda_tunable", ([&] {
        max_pool3d_tunable_kernel<scalar_t><<<blocks, threads, shared_mem_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            return_indices ? indices.data_ptr<int64_t>() : nullptr,
            batch_size, channels,
            input_d, input_h, input_w,
            output_d, output_h, output_w,
            kernel_size, stride, padding, dilation);
    }));

    if (return_indices) {
        return torch::stack({output, indices}, 0);
    }
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool3d_cuda_forward, "Tunable Block Size Max Pool 3D forward (CUDA)");
}
