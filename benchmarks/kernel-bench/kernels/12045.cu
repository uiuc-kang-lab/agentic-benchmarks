#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void balanced_hinge_loss_kernel(const float* __restrict__ predictions, const float* __restrict__ targets, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float sum = 0.0f;
    for (int i = idx; i < n; i += stride) {
        const float pred = __ldg(&predictions[i]);
        const float target = __ldg(&targets[i]);
        sum += fmaxf(0.0f, 1.0f - pred * target);
    }

    atomicAdd(output, sum);
}

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    CHECK_INPUT(predictions);
    CHECK_INPUT(targets);

    int n = predictions.numel();
    torch::Tensor output = torch::zeros({1}, predictions.options());

    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    balanced_hinge_loss_kernel<<<blocks, threads>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    // Compute the mean of the output tensor
    auto mean = output / n;
    return mean;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Balanced Workload Hinge Loss Forward");
}