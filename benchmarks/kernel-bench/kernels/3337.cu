#include "hip/hip_runtime.h"
#include <torch/extension.h>

// CUDA kernel that evenly partitions the input among threads
__global__ void balanced_swish_kernel(const float* __restrict__ x, float* __restrict__ y, int64_t n) {
    // Total number of threads in the grid
    int total_threads = gridDim.x * blockDim.x;
    // Global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Compute the number of elements per thread (rounded up)
    int chunk = (n + total_threads - 1) / total_threads;
    
    // Determine the start and end index for this thread
    int start = tid * chunk;
    int end = start + chunk;
    if (end > n) end = n;
    
    // Process a contiguous block of elements
    for (int i = start; i < end; i++) {
        float val = x[i];
        float sigmoid = 1.0f / (1.0f + expf(-val));
        y[i] = val * sigmoid;
    }
}

// PyTorch binding for the forward pass
torch::Tensor swish_forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    auto y = torch::empty_like(x);
    int64_t n = x.numel();

    // Choose grid dimensions for high occupancy on H100
    const int threads = 256;
    const int blocks = 1024;

    balanced_swish_kernel<<<blocks, threads>>>(x.data_ptr<float>(), y.data_ptr<float>(), n);
    
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &swish_forward, "Balanced Swish activation forward pass (CUDA)");
}
