#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void softplus_kernel_shared(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    extern __shared__ scalar_t shared_input[];
    
    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    if (idx < size) {
        shared_input[tid] = input[idx];
    }
    __syncthreads();
    
    // Process data from shared memory
    if (idx < size) {
        const scalar_t x = shared_input[tid];
        if (x > 20.0) {
            output[idx] = x;
        } else if (x < -20.0) {
            output[idx] = exp(x);
        } else {
            output[idx] = log1p(exp(x));
        }
    }
}

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    const int threads = 256;
    const int blocks = (size + threads - 1) / threads;
    const int shared_memory_size = threads * sizeof(float);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "softplus_forward_cuda", ([&] {
        softplus_kernel_shared<scalar_t><<<blocks, threads, shared_memory_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}