#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Device function to calculate input indices
__device__ __forceinline__ bool calculate_input_idx(
    int h_out, int w_out,
    int k_h, int k_w,
    int stride_h, int stride_w,
    int padding_h, int padding_w,
    int dilation_h, int dilation_w,
    int H_in, int W_in,
    int& h_in, int& w_in
) {
    h_in = h_out * stride_h - padding_h + k_h * dilation_h;
    w_in = w_out * stride_w - padding_w + k_w * dilation_w;
    return (h_in >= 0 && h_in < H_in && w_in >= 0 && w_in < W_in);
}

// Device function to calculate output index
__device__ __forceinline__ int calculate_output_idx(
    int n, int c_out, int h_out, int w_out,
    int C_out, int H_out, int W_out
) {
    return ((n * C_out + c_out) * H_out + h_out) * W_out + w_out;
}

// Device function to calculate weight index
__device__ __forceinline__ int calculate_weight_idx(
    int c_out, int c_in, int c_in_start,
    int k_h, int k_w, int K_h, int K_w,
    int C_in, int groups
) {
    return (((c_out * (C_in / groups) + (c_in - c_in_start)) * K_h + k_h) * K_w) + k_w;
}

// Main convolution kernel
__global__ void conv2d_cuda_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int N, int C_in, int H_in, int W_in,
    int C_out, int H_out, int W_out,
    int K_h, int K_w,
    int stride_h, int stride_w,
    int padding_h, int padding_w,
    int dilation_h, int dilation_w,
    int groups
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N * C_out * H_out * W_out) return;

    // Calculate output position
    const int w_out = tid % W_out;
    int tmp = tid / W_out;
    const int h_out = tmp % H_out;
    tmp = tmp / H_out;
    const int c_out = tmp % C_out;
    const int n = tmp / C_out;

    // Initialize output value with bias if present
    float value = (bias != nullptr) ? bias[c_out] : 0.0f;

    // Calculate group information
    const int group = c_out / (C_out / groups);
    const int c_in_start = group * (C_in / groups);
    const int c_in_end = c_in_start + (C_in / groups);

    // Main convolution loop with register optimization
    float sum = 0.0f;  // Use local accumulator to reduce register pressure
    const int c_in_offset = n * C_in;
    const int h_in_offset = H_in * W_in;
    const int w_in_offset = W_in;
    
    #pragma unroll 2  // Reduced unroll factor to decrease register pressure
    for (int c_in = c_in_start; c_in < c_in_end; ++c_in) {
        const int cin_idx = c_in - c_in_start;
        const int weight_base = ((c_out * (C_in / groups) + cin_idx) * K_h) * K_w;
        
        for (int k_h = 0; k_h < K_h; ++k_h) {
            // Pre-calculate h_in to avoid redundant computation
            const int h_in_temp = h_out * stride_h - padding_h + k_h * dilation_h;
            if (h_in_temp >= 0 && h_in_temp < H_in) {
                
                for (int k_w = 0; k_w < K_w; ++k_w) {
                    const int w_in_temp = w_out * stride_w - padding_w + k_w * dilation_w;
                    if (w_in_temp >= 0 && w_in_temp < W_in) {
                        const int input_idx = (c_in_offset + c_in) * h_in_offset + 
                                            h_in_temp * w_in_offset + w_in_temp;
                        const int weight_idx = weight_base + k_h * K_w + k_w;
                        sum += input[input_idx] * weight[weight_idx];
                    }
                }
            }
        }
    }
    value += sum;

    // Write output
    const int output_idx = calculate_output_idx(n, c_out, h_out, w_out, C_out, H_out, W_out);
    output[output_idx] = value;
}

torch::Tensor conv2d_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias_opt,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> dilation,
    int64_t groups
) {
    input = input.contiguous();
    weight = weight.contiguous();

    TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA");

    const int64_t N = input.size(0);
    const int64_t C_in = input.size(1);
    const int64_t H_in = input.size(2);
    const int64_t W_in = input.size(3);
    const int64_t C_out = weight.size(0);
    const int64_t K_h = weight.size(2);
    const int64_t K_w = weight.size(3);

    const int64_t H_out = (H_in + 2 * padding[0] - dilation[0] * (K_h - 1) - 1) / stride[0] + 1;
    const int64_t W_out = (W_in + 2 * padding[1] - dilation[1] * (K_w - 1) - 1) / stride[1] + 1;

    auto output = torch::zeros({N, C_out, H_out, W_out}, input.options());

    const int threads = 256;
    const int blocks = (N * C_out * H_out * W_out + threads - 1) / threads;

    conv2d_cuda_kernel<<<blocks, threads>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_opt.has_value() ? bias_opt.value().data_ptr<float>() : nullptr,
        output.data_ptr<float>(),
        N, C_in, H_in, W_in,
        C_out, H_out, W_out,
        K_h, K_w,
        stride[0], stride[1],
        padding[0], padding[1],
        dilation[0], dilation[1],
        groups
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv2d_cuda, "Modular 2D convolution (CUDA)",
        py::arg("input"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = std::vector<int64_t>{1, 1},
        py::arg("padding") = std::vector<int64_t>{0, 0},
        py::arg("dilation") = std::vector<int64_t>{1, 1},
        py::arg("groups") = 1
    );
}