#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void l2_normalize_kernel_optimized(
    const scalar_t* input,
    scalar_t* output,
    const int C,
    const int total_vectors,
    const int stride_C,
    const int outer_stride) {

    const int vector_idx = blockIdx.x;
    if (vector_idx >= total_vectors) return;

    const int base_offset = vector_idx * outer_stride;

    scalar_t sum = 0.0;

    // Compute sum of squares
    for (int k = threadIdx.x; k < C; k += blockDim.x) {
        const scalar_t val = input[base_offset + k * stride_C];
        sum += val * val;
    }

    // Block-wise reduction using shared memory
    __shared__ scalar_t shared_sum[256];
    shared_sum[threadIdx.x] = sum;
    __syncthreads();

    // Reduce within block using balanced binary tree
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Normalize and write output
    if (threadIdx.x == 0) {
        const scalar_t inv_norm = 1.0 / (sqrt(shared_sum[0]) + 1e-12);
        for (int k = 0; k < C; ++k) {
            output[base_offset + k * stride_C] = input[base_offset + k * stride_C] * inv_norm;
        }
    }
}

torch::Tensor forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(input.dim() >= 1, "Input must have at least 1 dimension");

    const int C = input.size(1);
    const int total_vectors = input.numel() / C;
    const int stride_C = input.stride(1);
    const int outer_stride = input.stride(0); // Simplified assumption for contiguous tensors

    auto output = torch::empty_like(input);

    const int threads = 256;
    const int blocks = (total_vectors + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "l2_normalize", [&] {
        l2_normalize_kernel_optimized<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            C,
            total_vectors,
            stride_C,
            outer_stride
        );
    });

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "L2 normalization along dim=1");
}