#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef TILE_SIZE
#define TILE_SIZE 32
#endif

// CUDA kernel for lower-triangular matrix multiplication with manually unrolled inner loop
__global__ void triangular_mm_kernel_unrolled(const float* __restrict__ A,
                                                const float* __restrict__ B,
                                                float* __restrict__ C,
                                                int N) {
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    if (row < N && col < N) {
        // For upper triangular elements, output zero
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            float sum = 0.f;
            int total_iters = row - col + 1;  // number of elements to sum
            int k = col;
            int unroll_iters = total_iters / 4;
            int remainder = total_iters % 4;

            // Unroll the loop by a factor of 4
            #pragma unroll
            for (int i = 0; i < unroll_iters; ++i) {
                sum += A[row * N + k]     * B[k * N + col];
                sum += A[row * N + k + 1] * B[(k + 1) * N + col];
                sum += A[row * N + k + 2] * B[(k + 2) * N + col];
                sum += A[row * N + k + 3] * B[(k + 3) * N + col];
                k += 4;
            }

            // Process any remaining iterations
            #pragma unroll
            for (int i = 0; i < remainder; ++i) {
                sum += A[row * N + k] * B[k * N + col];
                k++;
            }

            C[row * N + col] = sum;
        }
    }
}

// C++ interface exposed to PyTorch
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    triangular_mm_kernel_unrolled<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Unrolled triangular matrix multiplication (CUDA)");
}
