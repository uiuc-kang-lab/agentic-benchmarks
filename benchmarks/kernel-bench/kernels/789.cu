#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <iostream>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void matmul_kernel(float* A, float* B, float* C, int M, int N, int K) {
    __shared__ float shared_A[16][16];
    __shared__ float shared_B[16][16];
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; ++k) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matmul_kernel<<<numBlocks, threadsPerBlock>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, N, K);

    hipDeviceSynchronize();

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication (CUDA)");
}