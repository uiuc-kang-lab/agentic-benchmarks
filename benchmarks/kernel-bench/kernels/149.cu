#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
#define TILE_SIZE 16

// Constant memory for matrix dimensions
__constant__ int d_M;
__constant__ int d_N;
__constant__ int d_K;

#define CHECK_INPUT(x) \
    TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor"); \
    TORCH_CHECK(x.is_contiguous(), #x " must be contiguous"); \
    TORCH_CHECK(x.scalar_type() == torch::kFloat, #x " must be a float tensor");

__global__ void matrix_multiply_kernel(const float* __restrict__ A, 
                                     const float* __restrict__ B, 
                                     float* __restrict__ C) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    
    float sum = 0.0f;
    
    // Using constant memory dimensions
    for (int t = 0; t < (d_K - 1) / TILE_SIZE + 1; ++t) {
        if (row < d_M && t * TILE_SIZE + tx < d_K)
            As[ty][tx] = A[row * d_K + t * TILE_SIZE + tx];
        else
            As[ty][tx] = 0.0f;
            
        if (t * TILE_SIZE + ty < d_K && col < d_N)
            Bs[ty][tx] = B[(t * TILE_SIZE + ty) * d_N + col];
        else
            Bs[ty][tx] = 0.0f;
            
        __syncthreads();
        
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[ty][k] * Bs[k][tx];
        }
        __syncthreads();
    }
    
    if (row < d_M && col < d_N)
        C[row * d_N + col] = sum;
}

void matrix_multiply_cuda(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    // Copy dimensions to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_M), &M, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_N), &N, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_K), &K, sizeof(int));

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE,
                   (M + TILE_SIZE - 1) / TILE_SIZE);

    matrix_multiply_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>()
    );
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());
    matrix_multiply_cuda(A, B, C);
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication (CUDA)");
}