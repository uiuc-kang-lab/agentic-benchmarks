#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t, int STATIC_KERNEL_SIZE=-1>
__global__ void optimized_max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    extern __shared__ char shared_memory[];
    scalar_t* shared_input = reinterpret_cast<scalar_t*>(shared_memory);

    const int tid = threadIdx.x;
    const int output_idx = blockIdx.x * blockDim.x + tid;
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    const int ow = output_idx % output_width;
    const int oh = (output_idx / output_width) % output_height;
    const int c = (output_idx / (output_width * output_height)) % channels;
    const int b = output_idx / (output_width * output_height * channels);

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    if constexpr (STATIC_KERNEL_SIZE > 0) {
        #pragma unroll
        for (int kh = 0; kh < STATIC_KERNEL_SIZE; kh++) {
            #pragma unroll
            for (int kw = 0; kw < STATIC_KERNEL_SIZE; kw++) {
                const int ih = oh * stride - padding + kh * dilation;
                const int iw = ow * stride - padding + kw * dilation;

                if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
                    const int input_idx = b * (channels * input_height * input_width) +
                                        c * (input_height * input_width) +
                                        ih * input_width +
                                        iw;
                    max_val = max(max_val, input[input_idx]);
                }
            }
        }
    } else {
        for (int kh = 0; kh < kernel_size; kh++) {
            for (int kw = 0; kw < kernel_size; kw++) {
                const int ih = oh * stride - padding + kh * dilation;
                const int iw = ow * stride - padding + kw * dilation;

                if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
                    const int input_idx = b * (channels * input_height * input_width) +
                                        c * (input_height * input_width) +
                                        ih * input_width +
                                        iw;
                    max_val = max(max_val, input[input_idx]);
                }
            }
        }
    }

    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward_optimized(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool2d_cuda_forward_optimized", ([&] {
        switch(kernel_size) {
            case 2:
                optimized_max_pool2d_kernel<scalar_t, 2><<<blocks, threads>>>(
                    input.data_ptr<scalar_t>(),
                    output.data_ptr<scalar_t>(),
                    batch_size, channels, input_height, input_width,
                    output_height, output_width, kernel_size,
                    stride, padding, dilation
                );
                break;
            case 3:
                optimized_max_pool2d_kernel<scalar_t, 3><<<blocks, threads>>>(
                    input.data_ptr<scalar_t>(),
                    output.data_ptr<scalar_t>(),
                    batch_size, channels, input_height, input_width,
                    output_height, output_width, kernel_size,
                    stride, padding, dilation
                );
                break;
            default:
                optimized_max_pool2d_kernel<scalar_t, -1><<<blocks, threads>>>(
                    input.data_ptr<scalar_t>(),
                    output.data_ptr<scalar_t>(),
                    batch_size, channels, input_height, input_width,
                    output_height, output_width, kernel_size,
                    stride, padding, dilation
                );
        }
    }));

    return output;
}