#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>

namespace py = pybind11;

#define THREADS_PER_BLOCK 256
#define TILE_SIZE 16

template <typename scalar_t>
__global__ void depthwise_conv2d_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    int batch,
    int channels,
    int in_h, int in_w,
    int out_h, int out_w,
    int k,
    int stride,
    int padding,
    int dilation) {

  __shared__ scalar_t tile[TILE_SIZE][TILE_SIZE];
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch * channels * out_h * out_w;
  if (index >= total)
      return;

  int ow = index % out_w;
  int tmp = index / out_w;
  int oh = tmp % out_h;
  tmp = tmp / out_h;
  int c = tmp % channels;
  int n = tmp / channels;

  scalar_t sum = 0;
  
  // Load tile into shared memory only if needed
  int tile_row = threadIdx.x / TILE_SIZE;
  int tile_col = threadIdx.x % TILE_SIZE;
  if (tile_row < TILE_SIZE && tile_col < TILE_SIZE) {
    int ih = oh * stride - padding + tile_row;
    int iw = ow * stride - padding + tile_col;
    if (ih >= 0 && ih < in_h && iw >= 0 && iw < in_w) {
      tile[tile_row][tile_col] = input[n * channels * in_h * in_w + c * in_h * in_w + ih * in_w + iw];
    } else {
      tile[tile_row][tile_col] = 0;
    }
  }
  __syncthreads();

  for (int i = 0; i < k; ++i) {
    for (int j = 0; j < k; ++j) {
      int ih = oh * stride - padding + i * dilation;
      int iw = ow * stride - padding + j * dilation;
      if (ih >= 0 && ih < in_h && iw >= 0 && iw < in_w) {
        if (i < TILE_SIZE && j < TILE_SIZE) {
          sum += tile[i][j] * weight[c * k * k + i * k + j];
        } else {
          int input_idx = n * channels * in_h * in_w + c * in_h * in_w + ih * in_w + iw;
          sum += input[input_idx] * weight[c * k * k + i * k + j];
        }
      }
    }
  }
  
  if (bias != nullptr)
    sum += bias[c];
  output[index] = sum;
}

template <typename scalar_t>
__global__ void pointwise_conv2d_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    int batch,
    int in_channels,
    int out_channels,
    int h,
    int w) {

  __shared__ scalar_t shared_input[TILE_SIZE];
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch * out_channels * h * w;
  if (index >= total)
      return;

  int ow = index % w;
  int tmp = index / w;
  int oh = tmp % h;
  tmp = tmp / h;
  int oc = tmp % out_channels;
  int n = tmp / out_channels;

  scalar_t sum = 0;
  
  for (int ic_block = 0; ic_block < in_channels; ic_block += TILE_SIZE) {
    int remaining = min(TILE_SIZE, in_channels - ic_block);
    
    if (threadIdx.x < remaining) {
      int ic = ic_block + threadIdx.x;
      shared_input[threadIdx.x] = input[n * in_channels * h * w + ic * h * w + oh * w + ow];
    }
    __syncthreads();
    
    for (int i = 0; i < remaining; ++i) {
      sum += shared_input[i] * weight[oc * in_channels + (ic_block + i)];
    }
    __syncthreads();
  }
  
  if (bias != nullptr)
    sum += bias[oc];
  output[index] = sum;
}

torch::Tensor forward_cuda(
    const torch::Tensor& x,
    const torch::Tensor& depthwise_weight,
    const torch::Tensor& pointwise_weight,
    const torch::Tensor& depthwise_bias,
    const torch::Tensor& pointwise_bias,
    int stride,
    int padding,
    int dilation) {

  TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
  TORCH_CHECK(depthwise_weight.is_cuda(), "Depthwise weight must be a CUDA tensor");
  TORCH_CHECK(pointwise_weight.is_cuda(), "Pointwise weight must be a CUDA tensor");

  int batch = x.size(0);
  int in_channels = x.size(1);
  int in_h = x.size(2);
  int in_w = x.size(3);
  int k = depthwise_weight.size(2);
  int out_h = (in_h + 2 * padding - dilation * (k - 1) - 1) / stride + 1;
  int out_w = (in_w + 2 * padding - dilation * (k - 1) - 1) / stride + 1;

  auto depthwise_output = torch::empty({batch, in_channels, out_h, out_w}, x.options());
  int total_depthwise = batch * in_channels * out_h * out_w;
  int threads = THREADS_PER_BLOCK;
  int blocks = (total_depthwise + threads - 1) / threads;

  const void* depthwise_bias_ptr = (depthwise_bias.defined() && depthwise_bias.numel() > 0)
                                     ? depthwise_bias.data_ptr()
                                     : nullptr;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "depthwise_conv2d_cuda", ([&] {
    depthwise_conv2d_kernel<scalar_t><<<blocks, threads>>>(
        x.data_ptr<scalar_t>(),
        depthwise_weight.data_ptr<scalar_t>(),
        reinterpret_cast<const scalar_t*>(depthwise_bias_ptr),
        depthwise_output.data_ptr<scalar_t>(),
        batch, in_channels, in_h, in_w, out_h, out_w,
        k, stride, padding, dilation);
  }));

  int out_channels = pointwise_weight.size(0);
  auto output = torch::empty({batch, out_channels, out_h, out_w}, x.options());
  int total_pointwise = batch * out_channels * out_h * out_w;
  blocks = (total_pointwise + threads - 1) / threads;

  const void* pointwise_bias_ptr = (pointwise_bias.defined() && pointwise_bias.numel() > 0)
                                     ? pointwise_bias.data_ptr()
                                     : nullptr;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "pointwise_conv2d_cuda", ([&] {
    pointwise_conv2d_kernel<scalar_t><<<blocks, threads>>>(
        depthwise_output.data_ptr<scalar_t>(),
        pointwise_weight.data_ptr<scalar_t>(),
        reinterpret_cast<const scalar_t*>(pointwise_bias_ptr),
        output.data_ptr<scalar_t>(),
        batch, in_channels, out_channels, out_h, out_w);
  }));

  return output;
}

at::Tensor toTensor(const py::object& obj) {
  if (obj.is_none()) return at::Tensor();
  try {
    return obj.cast<at::Tensor>();
  } catch (const py::cast_error& e) {
    if (py::hasattr(obj, "data")) {
      return obj.attr("data").cast<at::Tensor>();
    }
    throw std::runtime_error("Expected a torch Tensor or Parameter.");
  }
}

at::Tensor forward_wrapper(py::object x_obj,
                           py::object depthwise_weight_obj,
                           py::object pointwise_weight_obj,
                           py::object depthwise_bias_obj,
                           py::object pointwise_bias_obj,
                           int stride,
                           int padding,
                           int dilation) {
  auto x = toTensor(x_obj);
  auto depthwise_weight = toTensor(depthwise_weight_obj);
  auto pointwise_weight = toTensor(pointwise_weight_obj);
  auto depthwise_bias = toTensor(depthwise_bias_obj);
  auto pointwise_bias = toTensor(pointwise_bias_obj);

  return forward_cuda(x, depthwise_weight, pointwise_weight,
                      depthwise_bias, pointwise_bias,
                      stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward_wrapper, "CUDA depthwise separable convolution forward");
}