#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <limits>

template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int ow = blockIdx.x * blockDim.x + threadIdx.x;
    const int oh = blockIdx.y * blockDim.y + threadIdx.y;
    const int bc = blockIdx.z;

    if (ow >= output_width || oh >= output_height) return;

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    #pragma unroll
    for (int kh = 0; kh < kernel_size; ++kh) {
        #pragma unroll
        for (int kw = 0; kw < kernel_size; ++kw) {
            const int ih = oh * stride - padding + kh * dilation;
            const int iw = ow * stride - padding + kw * dilation;
            
            if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
                const int input_idx = bc * input_height * input_width 
                                    + ih * input_width + iw;
                max_val = fmaxf(max_val, __ldg(&input[input_idx]));
            }
        }
    }

    output[bc * output_height * output_width + oh * output_width + ow] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const dim3 block(16, 16);
    const dim3 grid(
        (output_width + block.x - 1) / block.x,
        (output_height + block.y - 1) / block.y,
        batch_size * channels
    );

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool2d_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<grid, block>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size * channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}
