#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void diag_matmul_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    const int64_t N,
    const int64_t M
) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int stride = blockDim.x * gridDim.x;
    const int total = N * M;
    
    for (int idx = tid; idx < total; idx += stride) {
        const int row = idx / M;
        const int col = idx % M;
        C[idx] = A[row] * B[idx];
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.dim() == 1, "A must be a 1D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == B.size(0),
                "Dimension mismatch: A.size(0) must match B.size(0)");

    A = A.contiguous();
    B = B.contiguous();

    int64_t N = A.size(0);
    int64_t M = B.size(1);

    auto C = torch::empty({N, M}, B.options());

    const int threads = 128;
    const int max_blocks = 65535;
    const int blocks = min(max_blocks, (int)((N * M + threads - 1) / threads));

    diag_matmul_kernel<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N,
        M
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Diagonal matrix multiplication of A and B on the GPU");
}