#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAException.h>

#define BLOCK_SIZE 32

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CHECK_FLOAT(x) TORCH_CHECK(x.scalar_type() == torch::kFloat32, #x " must be a float32 tensor")

// This kernel minimizes warp divergence by refactoring boundary checks into uniform control flow
// for each tile load and store. For each tile iteration, the kernel tests once whether the entire
// tile lies within the valid matrix boundaries. If true, threads perform unconditional loads/stores, 
// otherwise they use conditional (but uniform) logic to avoid divergent branching within warps.

__global__ void uniform_control_tiled_matmul_kernel(const float* __restrict__ A,
                                                      const float* __restrict__ B,
                                                      float* __restrict__ C,
                                                      int N) {
    // Shared memory tiles for A and B
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE+1];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Each block computes a 32x32 output tile using a 16x16 thread block.
    // Each thread computes a 2x2 submatrix.
    int tx = threadIdx.x;  // 0..15
    int ty = threadIdx.y;  // 0..15

    // Compute starting indices for the 2x2 block computed by this thread
    int row = blockIdx.y * BLOCK_SIZE + ty * 2;
    int col = blockIdx.x * BLOCK_SIZE + tx * 2;

    // Registers to accumulate results
    float c00 = 0.0f, c01 = 0.0f, c10 = 0.0f, c11 = 0.0f;

    int threadsPerBlock = blockDim.x * blockDim.y; // Should be 256
    int index = threadIdx.y * blockDim.x + threadIdx.x;

    // Loop over tiles in the K dimension
    for (int m = 0; m < N; m += BLOCK_SIZE) {
        // For each tile, check once whether the entire A tile is in bounds
        bool validTileA = ((blockIdx.y * BLOCK_SIZE + BLOCK_SIZE) <= N) && ((m + BLOCK_SIZE) <= N);
        bool validTileB = ((m + BLOCK_SIZE) <= N) && ((blockIdx.x * BLOCK_SIZE + BLOCK_SIZE) <= N);

        int numLoads = (BLOCK_SIZE * BLOCK_SIZE) / threadsPerBlock; // 256/256 = 1 if BLOCK_SIZE*BLOCK_SIZE==1024, but here 32*32=1024/256=4
        
        // Load tile of A into shared memory
        for (int i = 0; i < numLoads; i++) {
            int loadIndex = index + i * threadsPerBlock;
            int ar = loadIndex / BLOCK_SIZE;
            int ac = loadIndex % BLOCK_SIZE;
            int global_r = blockIdx.y * BLOCK_SIZE + ar;
            int global_c = m + ac;
            if (validTileA) {
                As[ar][ac] = A[global_r * N + global_c];
            } else {
                As[ar][ac] = (global_r < N && global_c < N) ? A[global_r * N + global_c] : 0.0f;
            }
        }

        // Load tile of B into shared memory
        for (int i = 0; i < numLoads; i++) {
            int loadIndex = index + i * threadsPerBlock;
            int br = loadIndex / BLOCK_SIZE;
            int bc = loadIndex % BLOCK_SIZE;
            int global_r = m + br;
            int global_c = blockIdx.x * BLOCK_SIZE + bc;
            if (validTileB) {
                Bs[br][bc] = B[global_r * N + global_c];
            } else {
                Bs[br][bc] = (global_r < N && global_c < N) ? B[global_r * N + global_c] : 0.0f;
            }
        }
        __syncthreads();

        // Compute the partial 2x2 product for this tile
        for (int k = 0; k < BLOCK_SIZE; k++) {
            float a0 = As[ty * 2][k];
            float a1 = As[ty * 2 + 1][k];
            float b0 = Bs[k][tx * 2];
            float b1 = Bs[k][tx * 2 + 1];
            
            c00 += a0 * b0;
            c01 += a0 * b1;
            c10 += a1 * b0;
            c11 += a1 * b1;
        }
        __syncthreads();
    }

    // For writing output, check if the entire 32x32 tile is in bounds
    bool validWrite = ((blockIdx.y * BLOCK_SIZE + BLOCK_SIZE) <= N) && ((blockIdx.x * BLOCK_SIZE + BLOCK_SIZE) <= N);
    if (validWrite) {
        C[row * N + col] = c00;
        C[row * N + col + 1] = c01;
        C[(row + 1) * N + col] = c10;
        C[(row + 1) * N + col + 1] = c11;
    } else {
        if (row < N && col < N)
            C[row * N + col] = c00;
        if (row < N && (col + 1) < N)
            C[row * N + col + 1] = c01;
        if ((row + 1) < N && col < N)
            C[(row + 1) * N + col] = c10;
        if ((row + 1) < N && (col + 1) < N)
            C[(row + 1) * N + col + 1] = c11;
    }
}


torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_FLOAT(A);
    CHECK_FLOAT(B);

    TORCH_CHECK(A.dim() == 2 && A.size(0) == A.size(1), "A must be a square matrix");
    TORCH_CHECK(B.dim() == 2 && B.size(0) == B.size(1), "B must be a square matrix");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be of the same size");

    int64_t N = A.size(0);
    auto C = torch::zeros({N, N}, A.options());

    const float* A_data = A.data_ptr<float>();
    const float* B_data = B.data_ptr<float>();
    float* C_data = C.data_ptr<float>();

    // Launch configuration: each block computes a 32x32 tile using 16x16 threads (each computing a 2x2 submatrix)
    dim3 threads(16, 16);
    dim3 blocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    uniform_control_tiled_matmul_kernel<<<blocks, threads>>>(A_data, B_data, C_data, N);
    C10_CUDA_CHECK(hipGetLastError());

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Uniform control flow matrix multiplication kernel (CUDA)");
}
