#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// CUDA kernel for 2D transposed convolution using stride loops
__global__ void conv_transpose2d_forward_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch_size * out_channels * out_height * out_width;

  for (int idx = index; idx < total; idx += blockDim.x * gridDim.x) {
    // Decode index into (b, o, out_h, out_w)
    int w_out = idx % out_width;
    int temp = idx / out_width;
    int h_out = temp % out_height;
    temp /= out_height;
    int o = temp % out_channels;
    int b = temp / out_channels;

    float out_val = bias[o]; // start with bias

    for (int c = 0; c < in_channels; ++c) {
      for (int p = 0; p < kernel_size; ++p) {
        int h_unscaled = h_out + padding - p * dilation;
        if (h_unscaled % stride != 0)
          continue;
        int h_in = h_unscaled / stride;
        if (h_in < 0 || h_in >= in_height)
          continue;
        for (int q = 0; q < kernel_size; ++q) {
          int w_unscaled = w_out + padding - q * dilation;
          if (w_unscaled % stride != 0)
            continue;
          int w_in = w_unscaled / stride;
          if (w_in < 0 || w_in >= in_width)
            continue;
          int input_idx = ((b * in_channels + c) * in_height + h_in) * in_width + w_in;
          int weight_idx = ((c * out_channels + o) * kernel_size + p) * kernel_size + q;
          out_val += input[input_idx] * weight[weight_idx];
        }
      }
    }

    int output_idx = ((b * out_channels + o) * out_height + h_out) * out_width + w_out;
    output[output_idx] = out_val;
  }
}

// CUDA launcher function
torch::Tensor conv_transpose2d_forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {
  // Get input dimensions
  int batch_size = input.size(0);
  int in_channels = input.size(1);
  int in_height = input.size(2);
  int in_width = input.size(3);

  // Weight tensor: [in_channels, out_channels, kernel_size, kernel_size]
  int out_channels = weight.size(1);
  int kernel_size = weight.size(2); // assume square kernel

  // Calculate output dimensions
  int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  int out_width  = (in_width - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;

  auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());

  int total_threads = batch_size * out_channels * out_height * out_width;
  int threads = 1024;
  int blocks = (total_threads + threads - 1) / threads;

  conv_transpose2d_forward_kernel<<<blocks, threads>>>(
      input.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      output.data_ptr<float>(),
      batch_size,
      in_channels,
      out_channels,
      in_height,
      in_width,
      kernel_size,
      out_height,
      out_width,
      stride,
      padding,
      dilation);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in conv_transpose2d_forward_kernel: %s\n", hipGetErrorString(err));
  }

  return output;
}

// Wrapper to handle optional bias
torch::Tensor conv_transpose2d_forward_wrapper(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,  // accepts None
    int stride,
    int padding,
    int dilation) {

  int out_channels = weight.size(1);
  torch::Tensor bias;
  if (bias_obj.is(pybind11::none())) {
    bias = torch::zeros({out_channels}, weight.options());
  } else {
    bias = bias_obj.cast<torch::Tensor>();
  }

  return conv_transpose2d_forward_cuda(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &conv_transpose2d_forward_wrapper,
        "ConvTranspose2d forward (CUDA) with stride loops",
        pybind11::arg("input"),
        pybind11::arg("weight"),
        pybind11::arg("bias"),
        pybind11::arg("stride"),
        pybind11::arg("padding"),
        pybind11::arg("dilation"));
}