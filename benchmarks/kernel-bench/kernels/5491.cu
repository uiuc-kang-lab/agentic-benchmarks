#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    // Compute indices with improved alignment
    const int ow = output_idx % output_width;
    const int oh = (output_idx / output_width) % output_height;
    const int c = (output_idx / (output_width * output_height)) % channels;
    const int b = output_idx / (output_width * output_height * channels);

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    // Pre-compute base input offset for better memory access pattern
    const int base_input_offset = b * (channels * input_height * input_width) +
                                 c * (input_height * input_width);

    #pragma unroll
    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = oh * stride - padding + kh * dilation;
        if (ih >= 0 && ih < input_height) {
            const int ih_offset = ih * input_width;
            
            #pragma unroll
            for (int kw = 0; kw < kernel_size; kw++) {
                const int iw = ow * stride - padding + kw * dilation;
                
                if (iw >= 0 && iw < input_width) {
                    const int input_idx = base_input_offset + ih_offset + iw;
                    // Use __ldg for read-only memory access
                    max_val = max(max_val, __ldg(input + input_idx));
                }
            }
        }
    }

    // Aligned write to global memory
    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    // Ensure 128-bit alignment for output tensor
    auto output = torch::empty({batch_size, channels, output_height, output_width}, 
                             input.options().align_to(16));

    // Optimize thread block size for better occupancy
    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}