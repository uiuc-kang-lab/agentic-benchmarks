#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// CUDA kernel for 2D transposed convolution (conv_transpose2d) forward with shared memory optimization.
__global__ void conv_transpose2d_forward_kernel_shared(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

  

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch_size * out_channels * out_height * out_width;
  if (index >= total)
    return;

  // Decode index into (b, o, out_h, out_w)
  int w_out = index % out_width;
  int temp = index / out_width;
  int h_out = temp % out_height;
  temp /= out_height;
  int o = temp % out_channels;
  int b = temp / out_channels;

  float out_val = bias[o]; // start with bias

  for (int c = 0; c < in_channels; ++c) {
    // Load weights into shared memory
    if (threadIdx.x < kernel_size * kernel_size) {
      int p = threadIdx.x / kernel_size;
      int q = threadIdx.x % kernel_size;
      shared_weight[threadIdx.x] = weight[((c * out_channels + o) * kernel_size + p) * kernel_size + q];
    }
    __syncthreads();

    for (int p = 0; p < kernel_size; ++p) {
      int h_unscaled = h_out + padding - p * dilation;
      if (h_unscaled % stride != 0)
        continue;
      int h_in = h_unscaled / stride;
      if (h_in < 0 || h_in >= in_height)
        continue;
      for (int q = 0; q < kernel_size; ++q) {
        int w_unscaled = w_out + padding - q * dilation;
        if (w_unscaled % stride != 0)
          continue;
        int w_in = w_unscaled / stride;
        if (w_in < 0 || w_in >= in_width)
          continue;
        int input_idx = ((b * in_channels + c) * in_height + h_in) * in_width + w_in;
        out_val += input[input_idx] * shared_weight[p * kernel_size + q];
      }
    }
    __syncthreads();
  }

  int output_idx = ((b * out_channels + o) * out_height + h_out) * out_width + w_out;
  output[output_idx] = out_val;
}

torch::Tensor conv_transpose2d_forward_cuda_shared(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {
  
  // Get input dimensions.
  int batch_size = input.size(0);
  int in_channels = input.size(1);
  int in_height = input.size(2);
  int in_width = input.size(3);
  
  // Weight tensor: [in_channels, out_channels, kernel_size, kernel_size]
  int out_channels = weight.size(1);
  int kernel_size = weight.size(2);  // assume square kernel
  
  // Calculate output dimensions.
  int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  int out_width  = (in_width - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  
  auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());
  
  int total_threads = batch_size * out_channels * out_height * out_width;
  int threads = 1024;
  int blocks = (total_threads + threads - 1) / threads;
  int shared_mem_size = kernel_size * kernel_size * sizeof(float);
  
  conv_transpose2d_forward_kernel_shared<<<blocks, threads, shared_mem_size>>>(
      input.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      output.data_ptr<float>(),
      batch_size,
      in_channels,
      out_channels,
      in_height,
      in_width,
      kernel_size,
      out_height,
      out_width,
      stride,
      padding,
      dilation);
  
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in conv_transpose2d_forward_kernel_shared: %s\n", hipGetErrorString(err));
  }
  
  return output;
}

// Wrapper function to handle the possibility that the bias is None.
// If bias is None, we create a zero bias tensor of shape [out_channels].
torch::Tensor conv_transpose2d_forward_wrapper_shared(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,  // using py::object to accept None
    int stride,
    int padding,
    int dilation) {
  
  int out_channels = weight.size(1);
  torch::Tensor bias;
  if (bias_obj.is(pybind11::none())) {
    bias = torch::zeros({out_channels}, weight.options());
  } else {
    bias = bias_obj.cast<torch::Tensor>();
  }
  
  return conv_transpose2d_forward_cuda_shared(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &conv_transpose2d_forward_wrapper_shared,
        "ConvTranspose2d forward with shared memory (CUDA)",
        pybind11::arg("input"),
        pybind11::arg("weight"),
        pybind11::arg("bias"),
        pybind11::arg("stride"),
        pybind11::arg("padding"),
        pybind11::arg("dilation"));
}