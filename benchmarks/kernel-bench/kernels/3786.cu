#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_softplus(const scalar_t x) {
    if (x > static_cast<scalar_t>(20.0)) {
        return x;
    } else if (x < static_cast<scalar_t>(-20.0)) {
        return exp(x);
    }
    return log1p(exp(x));
}

template <typename scalar_t>
__global__ void softplus_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    extern __shared__ char shared_memory[];
    scalar_t* shared_data = reinterpret_cast<scalar_t*>(shared_memory);
    
    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + tid;
    
    // Load data into shared memory
    if (idx < size) {
        shared_data[tid] = input[idx];
    }
    
    // Single synchronization point to ensure shared memory is loaded
    __syncthreads();
    
    // Process data from shared memory
    if (idx < size) {
        output[idx] = compute_softplus(shared_data[tid]);
    }
}

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    const int threads = 256;  // Reduced thread count for better occupancy
    const int blocks = (size + threads - 1) / threads;
    const int shared_memory_size = threads * sizeof(float);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "softplus_forward_cuda", ([&] {
        softplus_kernel<scalar_t><<<blocks, threads, shared_memory_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}