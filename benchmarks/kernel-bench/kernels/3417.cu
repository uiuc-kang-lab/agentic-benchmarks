#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Inline GELU function for float precision
__device__ inline float gelu_function(float x) {
    // GELU(x) = x * 0.5 * (1 + erf(x / sqrt(2)))
    return x * 0.5f * (1.0f + erff(x / 1.4142135623730951f));
}

// CUDA kernel using a stride loop to process elements beyond the thread count
__global__ void gelu_kernel_stride(const float* __restrict__ input,
                                      float* __restrict__ output,
                                      size_t numel) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
    // Each thread processes multiple elements by advancing with a fixed stride
    for (size_t i = idx; i < numel; i += stride) {
        output[i] = gelu_function(input[i]);
    }
}

// Forward function callable from Python
torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
    TORCH_CHECK(x.scalar_type() == torch::ScalarType::Float, "Only float32 is supported");
    
    auto output = torch::empty_like(x);
    size_t numel = x.numel();

    // Tune thread and block counts
    int threads = 256;
    int blocks = (numel + threads - 1) / threads;

    // Launch kernel with stride looping to cover all elements
    gelu_kernel_stride<<<blocks, threads>>>(x.data_ptr<float>(), output.data_ptr<float>(), numel);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "GELU activation forward (CUDA) using stride loops");
}
