#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

// Custom CUDA kernel for transposed convolution
__global__ void conv_transpose2d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    float* __restrict__ output,
    int chunkN,        // number of batches in this chunk
    int C_in, int H, int W,
    int C_out, int K,  // square kernel size
    int stride,
    int padding,
    int H_out, int W_out) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total = chunkN * C_in * H * W;
    if (tid >= total) return;

    int w_idx = tid % W;
    int tmp = tid / W;
    int h_idx = tmp % H;
    tmp = tmp / H;
    int c_in = tmp % C_in;
    int n = tmp / C_in;

    float in_val = input[tid];
    
    for (int ki = 0; ki < K; ++ki) {
        for (int kj = 0; kj < K; ++kj) {
            int out_i = h_idx * stride - padding + ki;
            int out_j = w_idx * stride - padding + kj;
            if (out_i < 0 || out_i >= H_out || out_j < 0 || out_j >= W_out) continue;
            
            for (int oc = 0; oc < C_out; ++oc) {
                int weight_idx = c_in * (C_out * K * K) + oc * (K * K) + ki * K + kj;
                float w_val = weight[weight_idx];
                
                int out_index = n * (C_out * H_out * W_out) + oc * (H_out * W_out) + out_i * W_out + out_j;
                atomicAdd(&output[out_index], in_val * w_val);
            }
        }
    }
}

__global__ void add_bias_kernel(
    float* output,
    const float* bias,
    int total_elements,
    int C_out,
    int H_out,
    int W_out) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= total_elements) return;
    int channel = (tid / (H_out * W_out)) % C_out;
    output[tid] += bias[channel];
}

// Forward function implementing transposed convolution with CUDA streams
// This function partitions the input batch into chunks processed concurrently on separate streams.
// It overlaps kernel execution and memory operations, thus reducing runtime while maintaining full precision.

torch::Tensor conv_transpose2d_forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int64_t stride,
    int64_t padding,
    int64_t output_padding,
    int64_t groups) {

    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA");
    TORCH_CHECK(x.is_contiguous(), "Input tensor must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "Weight tensor must be contiguous");
    if (bias.has_value()) {
        auto bias_val = bias.value();
        TORCH_CHECK(bias_val.is_cuda(), "Bias tensor must be on CUDA");
        TORCH_CHECK(bias_val.is_contiguous(), "Bias tensor must be contiguous");
    }

    auto x_sizes = x.sizes();
    int N = x_sizes[0];
    int C_in = x_sizes[1];
    int H = x_sizes[2];
    int W = x_sizes[3];
    
    auto w_sizes = weight.sizes();
    int C_out = w_sizes[1];
    int K = w_sizes[2];

    int H_out = (H - 1) * stride - 2 * padding + K + output_padding;
    int W_out = (W - 1) * stride - 2 * padding + K + output_padding;

    auto output = torch::zeros({N, C_out, H_out, W_out}, x.options());

    int nstreams = 4;
    std::vector<hipStream_t> streams(nstreams);
    for (int i = 0; i < nstreams; i++) {
        hipStreamCreate(&streams[i]);
    }

    int chunk = (N + nstreams - 1) / nstreams;

    int block_size = 256;
    const float* weight_ptr = weight.data_ptr<float>();

    for (int i = 0; i < nstreams; i++) {
        int start = i * chunk;
        int end = std::min(N, (i + 1) * chunk);
        int chunkN = end - start;
        if (chunkN <= 0) continue;

        int num_elements = chunkN * C_in * H * W;
        const float* x_ptr = x.data_ptr<float>() + start * C_in * H * W;
        float* out_ptr = output.data_ptr<float>() + start * C_out * H_out * W_out;

        int grid_size = (num_elements + block_size - 1) / block_size;

        hipMemcpyAsync(out_ptr, x_ptr, num_elements * sizeof(float), hipMemcpyDeviceToDevice, streams[i]);

        conv_transpose2d_kernel<<<grid_size, block_size, 0, streams[i]>>>(
            x_ptr,
            weight_ptr,
            out_ptr,
            chunkN, C_in, H, W,
            C_out, K,
            stride,
            padding,
            H_out, W_out
        );
    }

    for (int i = 0; i < nstreams; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    if (bias.has_value()) {
        auto bias_tensor = bias.value();
        int total_output = N * C_out * H_out * W_out;
        int block_bias = 256;
        int grid_bias = (total_output + block_bias - 1) / block_bias;
        add_bias_kernel<<<grid_bias, block_bias>>>(
            output.data_ptr<float>(),
            bias_tensor.data_ptr<float>(),
            total_output, C_out, H_out, W_out
        );
        hipDeviceSynchronize();
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward, "ConvTranspose2d forward with streams and memory overlap (CUDA)");
}
