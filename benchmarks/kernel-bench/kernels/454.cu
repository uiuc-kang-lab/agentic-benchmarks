#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32
#define BLOCK_SIZE 512
#define WARPS_PER_BLOCK (BLOCK_SIZE/WARP_SIZE)

// Device function for warp-level reduction
__device__ float warp_reduce_sum(float val) {
    #pragma unroll
    for (int offset = WARP_SIZE/2; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

// Device function for block-level reduction
__device__ float block_reduce_sum(float val) {
    static __shared__ float shared[WARPS_PER_BLOCK];
    int lane = threadIdx.x % WARP_SIZE;
    int warp_id = threadIdx.x / WARP_SIZE;

    val = warp_reduce_sum(val);

    if (lane == 0) shared[warp_id] = val;
    __syncthreads();

    val = (threadIdx.x < WARPS_PER_BLOCK) ? shared[lane] : 0;
    if (warp_id == 0) val = warp_reduce_sum(val);

    return val;
}

// Kernel for matrix-vector multiplication
template <typename scalar_t>
__global__ void matvec_mul_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ C,
    const int64_t M,
    const int64_t K)
{
    const int64_t row = blockIdx.x;
    const int64_t tid = threadIdx.x;

    scalar_t sum = 0;
    const scalar_t* row_ptr = A + row * K;

    #pragma unroll 4
    for (int64_t k = tid; k < K; k += BLOCK_SIZE) {
        sum += row_ptr[k] * B[k];
    }

    sum = block_reduce_sum(sum);

    if (tid == 0) {
        C[row] = sum;
    }
}

torch::Tensor matvec_mul_cuda(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    
    auto A_contig = A.contiguous();
    auto B_contig = B.contiguous();
    
    const int64_t M = A.size(0);
    const int64_t K = A.size(1);
    
    auto B_flat = B_contig.view({-1});
    auto C = torch::zeros({M}, A.options());
    
    dim3 threads(BLOCK_SIZE);
    dim3 blocks(M);
    
    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matvec_mul_cuda", ([&] {
        matvec_mul_kernel<scalar_t><<<blocks, threads>>>(
            A_contig.data_ptr<scalar_t>(),
            B_flat.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M,
            K
        );
    }));
    
    return C.view({M, 1});
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matvec_mul_cuda, "Matrix-Vector Multiplication (CUDA)");
}