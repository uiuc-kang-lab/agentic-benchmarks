#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32
#define MAX_MATRIX_DIM 8192

__constant__ int d_N;  // Matrix dimension in constant memory
__constant__ int d_num_tiles;  // Number of tiles needed for the computation

// CUDA kernel for matrix multiplication
__global__ void matmul_kernel(const float* __restrict__ A,
                              const float* __restrict__ B,
                              float* __restrict__ C) {
    __shared__ float s_A[TILE_SIZE][TILE_SIZE];
    __shared__ float s_B[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;
    
    float value = 0;

    for (int i = 0; i < d_num_tiles; ++i) {
        if (row < d_N && i * TILE_SIZE + tx < d_N)
            s_A[ty][tx] = A[row * d_N + i * TILE_SIZE + tx];
        else
            s_A[ty][tx] = 0.0f;

        if (col < d_N && i * TILE_SIZE + ty < d_N)
            s_B[ty][tx] = B[(i * TILE_SIZE + ty) * d_N + col];
        else
            s_B[ty][tx] = 0.0f;

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k)
            value = fmaf(s_A[ty][k], s_B[k][tx], value);

        __syncthreads();
    }

    if (row < d_N && col < d_N)
        C[row * d_N + col] = value;
}

// C++ interface
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    // Check that A and B are float tensors, 2D, square, on CUDA
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be of same size");
    TORCH_CHECK(A.size(0) <= MAX_MATRIX_DIM, "Matrix dimension exceeds maximum supported size");

    int N = A.size(0);
    int num_tiles = (N + TILE_SIZE - 1) / TILE_SIZE;

    // Copy constants to device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_N), &N, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_tiles), &num_tiles, sizeof(int));

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, A.get_device());
    auto C = torch::zeros({N, N}, options);

    // Launch the CUDA kernel
    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    matmul_kernel<<<blocks, threads>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>());

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix Multiplication with Constant Memory (CUDA)");
}