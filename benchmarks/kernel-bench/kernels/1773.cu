#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel to compute C = tril(A * B) for lower triangular matrices A and B.
__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                       const float* __restrict__ B,
                                       float* __restrict__ C,
                                       int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            float sum = 0.f;
            for (int k = col; k <= row; ++k) {
                sum += A[row * N + k] * B[k * N + col];
            }
            C[row * N + col] = sum;
        }
    }
}

// C++ interface exposed to PyTorch.
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threads - 1) / threads, (N + threads - 1) / threads);

    // Create CUDA streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // Asynchronously copy data to device
    hipMemcpyAsync(A.data_ptr<float>(), A.data_ptr<float>(), A.numel() * sizeof(float), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(B.data_ptr<float>(), B.data_ptr<float>(), B.numel() * sizeof(float), hipMemcpyHostToDevice, stream2);

    // Launch the CUDA kernel in stream1
    triangular_mm_kernel<<<numBlocks, threadsPerBlock, 0, stream1>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    // Asynchronously copy result back to host
    hipMemcpyAsync(C.data_ptr<float>(), C.data_ptr<float>(), C.numel() * sizeof(float), hipMemcpyDeviceToHost, stream1);

    // Synchronize streams
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // Destroy streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication with streams (CUDA)");
}