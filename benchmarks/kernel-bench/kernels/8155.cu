#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void conv_transpose2d_kernel_unrolled(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int out_channels,
    const int kernel_h,
    const int kernel_w,
    const int stride,
    const int padding,
    const int output_padding,
    const int groups,
    const int dilation,
    const int out_height,
    const int out_width
) {
    const int total_elements = batch_size * out_channels * out_height * out_width;
    
    #pragma unroll 2
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; 
         idx < total_elements; 
         idx += blockDim.x * gridDim.x) {
        
        // Unravel index
        const int ow = idx % out_width;
        const int oh = (idx / out_width) % out_height;
        const int oc = (idx / (out_width * out_height)) % out_channels;
        const int b = idx / (out_width * out_height * out_channels);

        const int out_channels_per_group = out_channels / groups;
        const int g = oc / out_channels_per_group;
        const int oc_group = oc % out_channels_per_group;
        const int in_channels_per_group = in_channels / groups;
        const int ic_start = g * in_channels_per_group;

        scalar_t val = (bias != nullptr) ? bias[oc] : static_cast<scalar_t>(0);

        // Manual partial unroll of kernel height loop for common kernel sizes
        #pragma unroll 4
        for (int kh = 0; kh < kernel_h; ++kh) {
            const int h_in_temp = oh - kh * dilation + padding;
            if (h_in_temp % stride != 0) continue;
            
            const int h_in = h_in_temp / stride;
            if (h_in < 0 || h_in >= in_height) continue;

            // Manual partial unroll of kernel width loop
            #pragma unroll 4
            for (int kw = 0; kw < kernel_w; ++kw) {
                const int w_in_temp = ow - kw * dilation + padding;
                if (w_in_temp % stride != 0) continue;
                
                const int w_in = w_in_temp / stride;
                if (w_in < 0 || w_in >= in_width) continue;

                // Unroll the channel loop for better instruction-level parallelism
                #pragma unroll 4
                for (int ic = 0; ic < in_channels_per_group; ++ic) {
                    const int input_idx = b * (in_channels * in_height * in_width) +
                                        (ic_start + ic) * (in_height * in_width) +
                                        h_in * in_width + w_in;

                    const int weight_idx = (ic_start + ic) * (out_channels_per_group * kernel_h * kernel_w) +
                                         oc_group * (kernel_h * kernel_w) +
                                         kh * kernel_w + kw;

                    val += input[input_idx] * weight[weight_idx];
                }
            }
        }
        
        output[idx] = val;
    }
}

torch::Tensor forward(
    torch::Tensor input,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int output_padding,
    int groups,
    int dilation = 1
) {
    TORCH_CHECK(input.device().is_cuda(), "input must be a CUDA tensor");
    TORCH_CHECK(input.dim() == 4, "Input must be 4D");
    TORCH_CHECK(weight.dim() == 4, "Weight must be 4D");

    const int batch_size = input.size(0);
    const int in_channels = input.size(1);
    const int in_height = input.size(2);
    const int in_width = input.size(3);
    const int out_channels = weight.size(1) * groups;
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);

    if (bias.has_value() && bias->defined()) {
        TORCH_CHECK(bias->numel() == out_channels, "Bias must have out_channels elements");
        TORCH_CHECK(bias->device().is_cuda(), "Bias must be a CUDA tensor");
    }

    const int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_h - 1) + output_padding + 1;
    const int out_width = (in_width - 1) * stride - 2 * padding + dilation * (kernel_w - 1) + output_padding + 1;

    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());

    const int total_elements = output.numel();
    constexpr int THREADS_PER_BLOCK = 256;
    const int num_blocks = (total_elements + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "conv_transpose2d_cuda_unrolled", ([&] {
        conv_transpose2d_kernel_unrolled<scalar_t><<<num_blocks, THREADS_PER_BLOCK>>>(
            input.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            (bias.has_value() && bias->defined()) ? bias->data_ptr<scalar_t>() : nullptr,
            output.data_ptr<scalar_t>(),
            batch_size,
            in_channels,
            in_height,
            in_width,
            out_channels,
            kernel_h,
            kernel_w,
            stride,
            padding,
            output_padding,
            groups,
            dilation,
            out_height,
            out_width
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Unrolled Transposed 2D Convolution (CUDA)",
          py::arg("input"), py::arg("weight"), py::arg("bias") = py::none(),
          py::arg("stride"), py::arg("padding"), py::arg("output_padding"),
          py::arg("groups"), py::arg("dilation") = 1);
}