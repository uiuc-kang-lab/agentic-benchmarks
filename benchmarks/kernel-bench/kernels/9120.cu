#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <vector>

namespace py = pybind11;

// Constant memory for weights
__constant__ float c_weight[16384];

// Balanced kernel with even workload distribution

__global__ void conv_transpose2d_forward_kernel(
    const float* __restrict__ input,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int N,
    const int C_in,
    const int H_in,
    const int W_in,
    const int C_out,
    const int H_out,
    const int W_out,
    const int kH,
    const int kW,
    const int sH,
    const int sW,
    const int pH,
    const int pW
) {
    extern __shared__ float shared_input[];

    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    const int ow = bx % W_out;
    const int oh = (bx / W_out) % H_out;
    const int oc = (bx / (W_out * H_out)) % C_out;
    const int n = bx / (W_out * H_out * C_out);

    float sum = 0.0f;

    for (int ic = tx; ic < C_in; ic += blockDim.x) {
        for (int kh = 0; kh < kH; ++kh) {
            for (int kw = 0; kw < kW; ++kw) {
                const int i_val = oh + pH - kh;
                const int j_val = ow + pW - kw;

                if ((i_val % sH == 0) && (j_val % sW == 0)) {
                    const int i_in = i_val / sH;
                    const int j_in = j_val / sW;

                    if (i_in >= 0 && i_in < H_in && j_in >= 0 && j_in < W_in) {
                        const int input_idx = ((n * C_in + ic) * H_in + i_in) * W_in + j_in;
                        const int s_index = ((i_in * W_in) + j_in);
                        shared_input[s_index] = input[input_idx];

                        __syncthreads();

                        const int weight_idx = ((ic * C_out + oc) * kH + kh) * kW + kw;
                        sum += shared_input[s_index] * c_weight[weight_idx];
                    }
                }
            }
        }
    }

    if (bias != nullptr) {
        sum += bias[oc];
    }

    output[bx] = sum;
}

torch::Tensor conv_transpose2d_forward(
    torch::Tensor x,
    torch::Tensor weight,
    py::object bias_obj,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding
) {
    const int weight_size = weight.numel() * sizeof(float);
    if (weight_size > 64 * 1024) {
        // Fallback to cuDNN for large weights
        c10::optional<torch::Tensor> bias = c10::nullopt;
        if (!bias_obj.is_none()) {
            bias = bias_obj.cast<torch::Tensor>();
        }
        return at::conv_transpose2d(x, weight, bias, stride, padding);
    }

    hipMemcpyToSymbol(HIP_SYMBOL(c_weight), weight.data_ptr<float>(), weight_size);

    torch::Tensor bias;
    const float* bias_ptr = nullptr;
    if (!bias_obj.is_none()) {
        bias = bias_obj.cast<torch::Tensor>();
        bias_ptr = bias.data_ptr<float>();
    }

    const int N = x.size(0);
    const int C_in = x.size(1);
    const int H_in = x.size(2);
    const int W_in = x.size(3);
    const int C_out = weight.size(1);
    const int kH = weight.size(2);
    const int kW = weight.size(3);
    const int sH = stride[0];
    const int sW = stride[1];
    const int pH = padding[0];
    const int pW = padding[1];

    const int H_out = (H_in - 1) * sH - 2 * pH + kH;
    const int W_out = (W_in - 1) * sW - 2 * pW + kW;

    auto output = torch::zeros({N, C_out, H_out, W_out}, x.options());

    const int total_elements = N * C_out * H_out * W_out;

    // Maximum number of threads per block
    int threads_per_block = 256;
    int num_blocks = total_elements;

    size_t shared_memory_size = H_in * W_in * sizeof(float);

    conv_transpose2d_forward_kernel<<<num_blocks, threads_per_block, shared_memory_size>>>(
        x.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        N, C_in, H_in, W_in,
        C_out, H_out, W_out,
        kH, kW,
        sH, sW,
        pH, pW
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward, "Conv Transpose 2D forward with workload balancing",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride"),
          py::arg("padding"));
}
