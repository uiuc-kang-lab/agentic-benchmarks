#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <vector>

namespace py = pybind11;

__constant__ float c_weight[16384];

#define TILE_SIZE_H 8
#define TILE_SIZE_W 8
#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

__global__ void conv_transpose2d_forward_kernel_tiled(
    const float* __restrict__ input,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int N,
    const int C_in,
    const int H_in,
    const int W_in,
    const int C_out,
    const int H_out,
    const int W_out,
    const int kH,
    const int kW,
    const int sH,
    const int sW,
    const int pH,
    const int pW
) {
    const int tile_start_h = blockIdx.y * TILE_SIZE_H;
    const int tile_start_w = blockIdx.x * TILE_SIZE_W;
    const int n = blockIdx.z / C_out;
    const int oc = blockIdx.z % C_out;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    float partial_sums[TILE_SIZE_H/2][TILE_SIZE_W/2] = {0.0f};

    #pragma unroll
    for (int th = 0; th < TILE_SIZE_H/2; th++) {
        #pragma unroll
        for (int tw = 0; tw < TILE_SIZE_W/2; tw++) {
            const int oh = tile_start_h + th * 2 + ty % 2;
            const int ow = tile_start_w + tw * 2 + tx % 2;

            if (oh < H_out && ow < W_out) {
                float sum = 0.0f;

                #pragma unroll
                for (int ic = 0; ic < C_in; ++ic) {
                    #pragma unroll
                    for (int kh = 0; kh < kH; ++kh) {
                        #pragma unroll
                        for (int kw = 0; kw < kW; ++kw) {
                            const int i_val = oh + pH - kh;
                            const int j_val = ow + pW - kw;

                            if ((i_val % sH == 0) && (j_val % sW == 0)) {
                                const int i_in = i_val / sH;
                                const int j_in = j_val / sW;

                                if (i_in >= 0 && i_in < H_in && j_in >= 0 && j_in < W_in) {
                                    const int input_idx = ((n * C_in + ic) * H_in + i_in) * W_in + j_in;
                                    const int weight_idx = ((ic * C_out + oc) * kH + kh) * kW + kw;
                                    sum += input[input_idx] * c_weight[weight_idx];
                                }
                            }
                        }
                    }
                }

                if (bias != nullptr) {
                    sum += bias[oc];
                }

                partial_sums[th][tw] = sum;
            }
        }
    }

    #pragma unroll
    for (int th = 0; th < TILE_SIZE_H/2; th++) {
        #pragma unroll
        for (int tw = 0; tw < TILE_SIZE_W/2; tw++) {
            const int oh = tile_start_h + th * 2 + ty % 2;
            const int ow = tile_start_w + tw * 2 + tx % 2;

            if (oh < H_out && ow < W_out) {
                const int output_idx = ((n * C_out + oc) * H_out + oh) * W_out + ow;
                output[output_idx] = partial_sums[th][tw];
            }
        }
    }
}

torch::Tensor conv_transpose2d_forward(
    torch::Tensor x,
    torch::Tensor weight,
    py::object bias_obj,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding
) {
    const int weight_size = weight.numel() * sizeof(float);
    if (weight_size > 64 * 1024) {
        c10::optional<torch::Tensor> bias = c10::nullopt;
        if (!bias_obj.is_none()) {
            bias = bias_obj.cast<torch::Tensor>();
        }
        return at::conv_transpose2d(x, weight, bias, stride, padding);
    }

    hipMemcpyToSymbol(HIP_SYMBOL(c_weight), weight.data_ptr<float>(), weight_size);

    torch::Tensor bias;
    const float* bias_ptr = nullptr;
    if (!bias_obj.is_none()) {
        bias = bias_obj.cast<torch::Tensor>();
        bias_ptr = bias.data_ptr<float>();
    }

    const int N = x.size(0);
    const int C_in = x.size(1);
    const int H_in = x.size(2);
    const int W_in = x.size(3);
    const int C_out = weight.size(1);
    const int kH = weight.size(2);
    const int kW = weight.size(3);
    const int sH = stride[0];
    const int sW = stride[1];
    const int pH = padding[0];
    const int pW = padding[1];

    const int H_out = (H_in - 1) * sH - 2 * pH + kH;
    const int W_out = (W_in - 1) * sW - 2 * pW + kW;

    auto output = torch::zeros({N, C_out, H_out, W_out}, x.options());

    dim3 block(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 grid(
        (W_out + TILE_SIZE_W - 1) / TILE_SIZE_W,
        (H_out + TILE_SIZE_H - 1) / TILE_SIZE_H,
        N * C_out
    );

    conv_transpose2d_forward_kernel_tiled<<<grid, block>>>(
        x.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        N, C_in, H_in, W_in,
        C_out, H_out, W_out,
        kH, kW, sH, sW, pH, pW
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward, "Conv Transpose 2D forward with tiled workload distribution",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride"),
          py::arg("padding"));
}