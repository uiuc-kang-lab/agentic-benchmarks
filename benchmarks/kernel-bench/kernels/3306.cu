#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void swish_kernel(const float* x, float* y, int64_t n) {
    const int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int64_t index = tid * 4;

    if (index + 3 < n) {
        float4 x4 = reinterpret_cast<const float4*>(x + index)[0];
        float4 y4;
        y4.x = x4.x * (1.0f / (1.0f + expf(-x4.x)));
        y4.y = x4.y * (1.0f / (1.0f + expf(-x4.y)));
        y4.z = x4.z * (1.0f / (1.0f + expf(-x4.z)));
        y4.w = x4.w * (1.0f / (1.0f + expf(-x4.w)));
        reinterpret_cast<float4*>(y + index)[0] = y4;
    } else {
        for (int i = 0; i < 4; ++i) {
            if (index + i < n) {
                float val = x[index + i];
                y[index + i] = val * (1.0f / (1.0f + expf(-val)));
            }
        }
    }
}

torch::Tensor swish_forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    auto y = torch::empty_like(x);
    const int64_t n = x.numel();
    const int threads = 256;
    const int blocks = (n + 4 * threads - 1) / (4 * threads);
    
    swish_kernel<<<blocks, threads>>>(x.data_ptr<float>(), y.data_ptr<float>(), n);
    
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &swish_forward, "Swish activation forward pass (CUDA)");
}