#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <iostream>

#define BLOCK_SIZE 16

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void matrixMulKernel(const float* A, const float* B, float* C, int M, int N, int K) {
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by * BLOCK_SIZE + ty;
    int col = bx * BLOCK_SIZE + tx;
    
    float sum = 0.0f;
    
    for (int i = 0; i < (K-1)/BLOCK_SIZE + 1; ++i) {
        if (row < M && i*BLOCK_SIZE + tx < K)
            As[ty][tx] = A[row*K + i*BLOCK_SIZE + tx];
        else
            As[ty][tx] = 0.0f;
            
        if (col < N && i*BLOCK_SIZE + ty < K)
            Bs[ty][tx] = B[(i*BLOCK_SIZE + ty)*N + col];
        else
            Bs[ty][tx] = 0.0f;
            
        __syncthreads();
        
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            sum += As[ty][k] * Bs[k][tx];
        }
        __syncthreads();
    }
    
    if (row < M && col < N) {
        C[row*N + col] = sum;
    }
}

void matrix_multiply_cuda(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    float *d_A = A.data_ptr<float>();
    float *d_B = B.data_ptr<float>();
    float *d_C = C.data_ptr<float>();

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
                   (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());
    matrix_multiply_cuda(A, B, C);
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication (CUDA)");
}