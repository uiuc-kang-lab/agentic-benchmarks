#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor matmul_cuda(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform matrix multiplication using cuBLAS
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B.data_ptr<float>(), N, A.data_ptr<float>(), K, &beta, C.data_ptr<float>(), N);

    // Destroy cuBLAS handle
    hipblasDestroy(handle);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_cuda, "Matrix multiplication (CUDA)");
}