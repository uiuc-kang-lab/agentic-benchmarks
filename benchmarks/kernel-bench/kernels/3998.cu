#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__device__ inline float fast_elu(float x, float alpha) {
    return (x > 0.0f) ? x : alpha * (__expf(x) - 1.0f); // Use __expf for fast computation
}

__global__ void elu_kernel_reduced_divergence(const float* __restrict__ x, float* __restrict__ out, float alpha, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    while (idx < n) {
        float val = x[idx];
        out[idx] = fast_elu(val, alpha);
        idx += stride;
    }
}

torch::Tensor elu_cuda(torch::Tensor x, float alpha) {
    CHECK_INPUT(x);
    auto out = torch::empty_like(x);
    int n = x.numel();

    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;

    elu_kernel_reduced_divergence<<<blocks, threads>>>(x.data_ptr<float>(), out.data_ptr<float>(), alpha, n);

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &elu_cuda, "ELU activation with reduced warp divergence (CUDA)");
}
