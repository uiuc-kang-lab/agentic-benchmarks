#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#define THREADS_PER_BLOCK 256
#define WARP_SIZE 32

__inline__ __device__ float warp_reduce_max(float val) {
    #pragma unroll
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
        val = max(val, __shfl_down_sync(0xffffffff, val, offset));
    }
    return val;
}

__inline__ __device__ float warp_reduce_sum(float val) {
    #pragma unroll
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__global__ void softmax_kernel(const float* __restrict__ x, float* __restrict__ y, int num_features) {
    int batch_idx = blockIdx.x;
    int tid = threadIdx.x;
    int lane_id = tid % WARP_SIZE;
    int warp_id = tid / WARP_SIZE;
    int num_warps = THREADS_PER_BLOCK / WARP_SIZE;

    const float* x_row = x + batch_idx * num_features;
    float* y_row = y + batch_idx * num_features;

    extern __shared__ float shared[];
    float* warp_maxes = shared;
    float* warp_sums = &shared[num_warps];

    // Find max value using thread-local variable first
    float thread_max = -INFINITY;
    for (int i = tid; i < num_features; i += THREADS_PER_BLOCK) {
        thread_max = max(thread_max, x_row[i]);
    }

    // Warp-level reduction for max
    float warp_max = warp_reduce_max(thread_max);

    // Store warp results in shared memory
    if (lane_id == 0) {
        warp_maxes[warp_id] = warp_max;
    }
    __syncthreads();

    // Final reduction across warps
    if (warp_id == 0 && lane_id < num_warps) {
        float final_max = warp_maxes[lane_id];
        final_max = warp_reduce_max(final_max);
        warp_maxes[0] = final_max;
    }
    __syncthreads();

    float max_val = warp_maxes[0];

    // Compute exponentials and partial sums
    float thread_sum = 0.0f;
    for (int i = tid; i < num_features; i += THREADS_PER_BLOCK) {
        float exp_val = __expf(x_row[i] - max_val);
        y_row[i] = exp_val;
        thread_sum += exp_val;
    }

    // Warp-level reduction for sum
    float warp_sum = warp_reduce_sum(thread_sum);

    // Store warp results
    if (lane_id == 0) {
        warp_sums[warp_id] = warp_sum;
    }
    __syncthreads();

    // Final reduction across warps
    if (warp_id == 0 && lane_id < num_warps) {
        float final_sum = warp_sums[lane_id];
        final_sum = warp_reduce_sum(final_sum);
        warp_sums[0] = final_sum;
    }
    __syncthreads();

    float sum_val = warp_sums[0];
    float inv_sum = 1.0f / sum_val;

    // Final normalization
    for (int i = tid; i < num_features; i += THREADS_PER_BLOCK) {
        y_row[i] *= inv_sum;
    }
}

void softmax_forward_cuda(const float* x, float* y, int batch_size, int num_features) {
    dim3 block_dim(THREADS_PER_BLOCK);
    dim3 grid_dim(batch_size);

    int num_warps = THREADS_PER_BLOCK / WARP_SIZE;
    int shared_mem_size = sizeof(float) * num_warps * 2;

    softmax_kernel<<<grid_dim, block_dim, shared_mem_size>>>(x, y, num_features);
}

torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor.");
    TORCH_CHECK(x.dim() == 2, "Input tensor must be 2D.");
    TORCH_CHECK(x.scalar_type() == torch::kFloat32, "Input tensor must be float32.");

    auto y = torch::empty_like(x);
    softmax_forward_cuda(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        x.size(0),
        x.size(1)
    );
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Softmax forward (CUDA)");
}