#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Modular device function to compute convolution for a single output element
__device__ __forceinline__ float compute_conv_at(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    int b,
    int c_out,
    int h_out,
    int w_out,
    int in_channels,
    int in_h,
    int in_w,
    int kernel_h,
    int kernel_w,
    int stride_h,
    int stride_w,
    int padding_h,
    int padding_w,
    int dilation_h,
    int dilation_w,
    int groups,
    int channels_per_group
) {
    int g = c_out / channels_per_group;
    int m = c_out % channels_per_group;
    float sum = 0.0f;
    for (int kh = 0; kh < kernel_h; ++kh) {
        for (int kw = 0; kw < kernel_w; ++kw) {
            int h_in = h_out * stride_h - padding_h + kh * dilation_h;
            int w_in = w_out * stride_w - padding_w + kw * dilation_w;
            if (h_in >= 0 && h_in < in_h && w_in >= 0 && w_in < in_w) {
                int input_idx = ((b * in_channels + g) * in_h + h_in) * in_w + w_in;
                int weight_idx = ((g * channels_per_group + m) * kernel_h + kh) * kernel_w + kw;
                sum += input[input_idx] * weight[weight_idx];
            }
        }
    }
    return sum;
}

// Kernel that computes depthwise convolution using the modular device function
__global__ void modular_depthwise_conv2d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int in_h,
    int in_w,
    int out_channels,
    int out_h,
    int out_w,
    int kernel_h,
    int kernel_w,
    int stride_h,
    int stride_w,
    int padding_h,
    int padding_w,
    int dilation_h,
    int dilation_w,
    int groups,
    int channels_per_group
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch_size * out_channels * out_h * out_w;
    if (idx >= total) return;

    // Decode flattened index into (b, c_out, h_out, w_out)
    int w = idx % out_w;
    idx /= out_w;
    int h = idx % out_h;
    idx /= out_h;
    int c = idx % out_channels;
    int b = idx / out_channels;

    float value = compute_conv_at(input, weight, b, c, h, w,
                                  in_channels, in_h, in_w,
                                  kernel_h, kernel_w,
                                  stride_h, stride_w,
                                  padding_h, padding_w,
                                  dilation_h, dilation_w,
                                  groups, channels_per_group);
    
    if (bias != nullptr) {
        value += bias[c];
    }

    output[((b * out_channels + c) * out_h + h) * out_w + w] = value;
}

// Host function to launch the kernel
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    int stride_h,
    int stride_w,
    int padding_h,
    int padding_w,
    int dilation_h,
    int dilation_w,
    int groups
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int in_h = x.size(2);
    int in_w = x.size(3);

    int kernel_h = weight.size(2);
    int kernel_w = weight.size(3);
    int out_channels = groups * weight.size(1);
    int channels_per_group = out_channels / groups;

    int out_h = (in_h + 2 * padding_h - dilation_h * (kernel_h - 1) - 1) / stride_h + 1;
    int out_w = (in_w + 2 * padding_w - dilation_w * (kernel_w - 1) - 1) / stride_w + 1;

    auto output = torch::empty({batch_size, out_channels, out_h, out_w}, x.options());

    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        bias_ptr = bias->data_ptr<float>();
    }

    int total_elements = batch_size * out_channels * out_h * out_w;
    int threads = 256;
    int blocks = (total_elements + threads - 1) / threads;

    modular_depthwise_conv2d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        in_h,
        in_w,
        out_channels,
        out_h,
        out_w,
        kernel_h,
        kernel_w,
        stride_h,
        stride_w,
        padding_h,
        padding_w,
        dilation_h,
        dilation_w,
        groups,
        channels_per_group
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Modular Depthwise Conv2D forward (CUDA)");
}
