#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void strided_batch_cumsum_kernel(
    const scalar_t* __restrict__ x,
    const bool* __restrict__ mask,
    scalar_t* __restrict__ output,
    int64_t N,
    int64_t L) {

    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    // Each thread processes multiple rows in a strided fashion
    for (int64_t row = tid; row < N; row += stride) {
        const scalar_t* x_row = x + row * L;
        const bool* mask_row = mask + row * L;
        scalar_t* output_row = output + row * L;
        
        // Process the row using registers for accumulation
        scalar_t sum = 0;
        
        // Use register array for better performance
        #pragma unroll 4
        for (int64_t i = 0; i < L; ++i) {
            scalar_t val = x_row[i];
            bool m = mask_row[i];
            // Avoid branch by using multiplication
            sum += val * static_cast<scalar_t>(m);
            output_row[i] = sum;
        }
    }
}

torch::Tensor masked_cumsum(
    const torch::Tensor& x,
    const torch::Tensor& mask,
    int64_t dim) {

    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(mask.is_cuda(), "mask must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(mask.is_contiguous(), "mask must be contiguous");
    TORCH_CHECK(x.sizes() == mask.sizes(), "x and mask must have the same shape");
    TORCH_CHECK(mask.scalar_type() == torch::kBool, "mask must be a boolean tensor");

    if (dim < 0) {
        dim += x.dim();
    }
    TORCH_CHECK(dim >= 0 && dim < x.dim(), "Invalid dimension");

    std::vector<int64_t> perm;
    for (int64_t i = 0; i < x.dim(); ++i) {
        if (i != dim)
            perm.push_back(i);
    }
    perm.push_back(dim);

    auto x_permuted = x.permute(perm).contiguous();
    auto mask_permuted = mask.permute(perm).contiguous();

    int64_t N = x_permuted.numel() / x_permuted.size(-1);
    int64_t L = x_permuted.size(-1);

    auto x_flat = x_permuted.view({N, L});
    auto mask_flat = mask_permuted.view({N, L});
    auto output_flat = torch::empty_like(x_flat);

    // Optimize thread and block configuration
    const int threads_per_block = 256;
    const int max_blocks = 1024;
    const int num_blocks = std::min(max_blocks, (int)((N + threads_per_block - 1) / threads_per_block));

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "masked_cumsum_cuda", ([&] {
        strided_batch_cumsum_kernel<scalar_t><<<num_blocks, threads_per_block>>>(
            x_flat.data_ptr<scalar_t>(),
            mask_flat.data_ptr<bool>(),
            output_flat.data_ptr<scalar_t>(),
            N,
            L
        );
    }));

    auto output_permuted = output_flat.view(x_permuted.sizes());
    std::vector<int64_t> inv_perm(perm.size());
    for (size_t i = 0; i < perm.size(); ++i) {
        inv_perm[perm[i]] = i;
    }
    return output_permuted.permute(inv_perm);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &masked_cumsum, "Masked Cumulative Sum (CUDA)");
}