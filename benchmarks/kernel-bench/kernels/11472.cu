#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Kernel using 2D grid and block indexing to map a flat 1D input onto a nearly square 2D domain
__global__ void kl_div_kernel_2d_indexing(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets,
    float* __restrict__ output,
    const int n,
    const int total_width) {

    // Compute global 2D thread coordinates
    int global_x = threadIdx.x + blockIdx.x * blockDim.x;
    int global_y = threadIdx.y + blockIdx.y * blockDim.y;
    // Flatten the 2D coordinate into a 1D index (row-major order)
    int idx = global_y * total_width + global_x;

    // Compute the stride: total number of threads covering one full pass of the data
    int stride = total_width * gridDim.y * blockDim.y;

    float local_sum = 0.0f;
    // Grid-stride loop: Each thread processes multiple elements
    for (; idx < n; idx += stride) {
        float lp = log_predictions[idx];
        float t  = targets[idx];
        local_sum += expf(lp) - t * lp;
    }

    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    
    // Use warp-level reduction with shuffle intrinsics to reduce shared memory accesses
    unsigned int mask = 0xffffffff; // full mask for 32 threads
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        local_sum += __shfl_down_sync(mask, local_sum, offset);
    }
    
    // Write each warp's result to shared memory
    extern __shared__ float shared[];
    if ((tid & (warpSize - 1)) == 0) {
        shared[tid >> 5] = local_sum;
    }
    __syncthreads();

    // Final reduction performed by the first warp
    int num_warps = (blockDim.x * blockDim.y + warpSize - 1) / warpSize;
    if (tid < num_warps) {
        local_sum = shared[tid];
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            local_sum += __shfl_down_sync(mask, local_sum, offset);
        }
        if (tid == 0) {
            atomicAdd(output, local_sum);
        }
    }
}

// CUDA forward function with 2D indexing optimization
torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {

    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());

    // Define 2D block dimensions (16x16 = 256 threads per block)
    const int block_x = 16;
    const int block_y = 16;
    dim3 block(block_x, block_y);

    // Map the flat input to a nearly square 2D domain
    int total_columns = ceilf(sqrtf((float)n));
    int total_rows = (n + total_columns - 1) / total_columns;

    // Determine grid dimensions based on block size
    int grid_x = (total_columns + block_x - 1) / block_x;
    int grid_y = (total_rows + block_y - 1) / block_y;
    dim3 grid(grid_x, grid_y);

    int shared_mem_size = block_x * block_y * sizeof(float);
    // total_width is the effective width of the 2D mapping
    int total_width = grid_x * block_x;

    kl_div_kernel_2d_indexing<<<grid, block, shared_mem_size>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n,
        total_width
    );

    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward with 2D indexing optimization (CUDA)");
}
