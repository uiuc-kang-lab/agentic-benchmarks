#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__device__ __forceinline__ void get_pool_indices(
    const int output_idx,
    const int output_width,
    const int output_height,
    const int channels,
    int& ow,
    int& oh,
    int& c,
    int& b
) {
    ow = output_idx % output_width;
    oh = (output_idx / output_width) % output_height;
    c = (output_idx / (output_width * output_height)) % channels;
    b = output_idx / (output_width * output_height * channels);
}

template <typename scalar_t>
__device__ __forceinline__ bool is_valid_input_index(
    const int ih,
    const int iw,
    const int input_height,
    const int input_width
) {
    return (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_max_value(
    const scalar_t* input,
    const int b,
    const int c,
    const int oh,
    const int ow,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation,
    const int input_height,
    const int input_width,
    const int channel_stride
) {
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    const int base_input_offset = b * channel_stride + c * (input_height * input_width);

    #pragma unroll
    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = oh * stride - padding + kh * dilation;
        
        #pragma unroll
        for (int kw = 0; kw < kernel_size; kw++) {
            const int iw = ow * stride - padding + kw * dilation;
            
            if (is_valid_input_index<scalar_t>(ih, iw, input_height, input_width)) {
                const int input_idx = base_input_offset + ih * input_width + iw;
                max_val = max(max_val, input[input_idx]);
            }
        }
    }
    return max_val;
}

template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    int ow, oh, c, b;
    get_pool_indices<scalar_t>(output_idx, output_width, output_height, channels, ow, oh, c, b);

    const int channel_stride = channels * input_height * input_width;
    output[output_idx] = compute_max_value<scalar_t>(
        input, b, c, oh, ow,
        kernel_size, stride, padding, dilation,
        input_height, input_width, channel_stride
    );
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}