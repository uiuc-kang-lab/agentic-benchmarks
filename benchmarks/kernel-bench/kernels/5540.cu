#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    // 3D grid mapping: x->width, y->height, z->(batch*channels)
    const int ow = blockIdx.x * blockDim.x + threadIdx.x;
    const int oh = blockIdx.y * blockDim.y + threadIdx.y;
    const int bc_idx = blockIdx.z * blockDim.z + threadIdx.z;
    
    const int c = bc_idx % channels;
    const int b = bc_idx / channels;
    
    if (ow >= output_width || oh >= output_height || b >= batch_size) return;

    const int input_stride_batch = channels * input_height * input_width;
    const int input_stride_channel = input_height * input_width;
    
    const int base_offset = b * input_stride_batch + c * input_stride_channel;
    const int ih_start = oh * stride - padding;
    const int iw_start = ow * stride - padding;

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    
    #pragma unroll 4
    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = ih_start + kh * dilation;
        if (ih >= 0 && ih < input_height) {
            const int h_offset = ih * input_width;
            #pragma unroll 4
            for (int kw = 0; kw < kernel_size; kw++) {
                const int iw = iw_start + kw * dilation;
                if (iw >= 0 && iw < input_width) {
                    const int input_idx = base_offset + h_offset + iw;
                    max_val = max(max_val, __ldg(&input[input_idx]));
                }
            }
        }
    }

    const int output_idx = b * (channels * output_height * output_width) +
                          c * (output_height * output_width) +
                          oh * output_width +
                          ow;
    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    // 3D thread block configuration
    const dim3 threads(8, 8, 4);
    const dim3 blocks(
        (output_width + threads.x - 1) / threads.x,
        (output_height + threads.y - 1) / threads.y,
        ((batch_size * channels) + threads.z - 1) / threads.z
    );

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}