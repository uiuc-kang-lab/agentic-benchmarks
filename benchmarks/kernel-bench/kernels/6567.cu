#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void simple_max_reduce_kernel(
    const scalar_t* input,
    scalar_t* output,
    const int64_t outer_size,
    const int64_t dim_size,
    const int64_t inner_size
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_elements = outer_size * inner_size;
    
    if (idx >= total_elements) return;
    
    const int outer_idx = idx / inner_size;
    const int inner_idx = idx % inner_size;
    const int64_t start_idx = outer_idx * dim_size * inner_size + inner_idx;
    
    scalar_t max_val = input[start_idx];
    for (int i = 1; i < dim_size; i++) {
        max_val = max(max_val, input[start_idx + i * inner_size]);
    }
    output[idx] = max_val;
}

template <typename scalar_t>
__global__ void parallel_max_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t dim_size,
    const int64_t inner_size,
    const int64_t num_outputs
) {
    for (int out_idx = blockIdx.x; out_idx < num_outputs; out_idx += gridDim.x) {
        int outer_idx = out_idx / inner_size;
        int inner_idx = out_idx % inner_size;
        int64_t base = outer_idx * dim_size * inner_size + inner_idx;

        int tid = threadIdx.x;
        int block_size = blockDim.x;
        
        scalar_t thread_max = input[base];
        for (int j = tid + 1; j < dim_size; j += block_size) {
            thread_max = max(thread_max, input[base + j * inner_size]);
        }
        
        extern __shared__ char sdata[];
        scalar_t* shmax = reinterpret_cast<scalar_t*>(sdata);
        shmax[tid] = thread_max;
        __syncthreads();

        for (unsigned int s = block_size / 2; s > 0; s >>= 1) {
            if (tid < s) {
                shmax[tid] = max(shmax[tid], shmax[tid + s]);
            }
            __syncthreads();
        }

        if (tid == 0) {
            output[out_idx] = shmax[0];
        }
    }
}

torch::Tensor max_reduce_cuda_forward(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();
    
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) outer_size *= input.size(i);
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < input.dim(); i++) inner_size *= input.size(i);

    const int64_t dim_size = input.size(dim);
    const int64_t num_outputs = outer_size * inner_size;
    
    auto output_sizes = input.sizes().vec();
    output_sizes.erase(output_sizes.begin() + dim);
    auto output = torch::empty(output_sizes, input.options());

    const bool use_parallel = (dim_size >= 512 || num_outputs <= 1024);
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "adaptive_max_reduce_forward", ([&] {
        if (use_parallel) {
            int threads = (dim_size < 256 ? dim_size : 256);
            int blocks = (num_outputs < 1024 ? num_outputs : 1024);
            size_t shm_size = threads * sizeof(scalar_t);
            
            parallel_max_reduce_kernel<scalar_t><<<blocks, threads, shm_size>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size,
                inner_size,
                num_outputs
            );
        } else {
            const int threads = 256;
            const int blocks = (num_outputs + threads - 1) / threads;
            
            simple_max_reduce_kernel<scalar_t><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                outer_size,
                dim_size,
                inner_size
            );
        }
    }));
    
    return output;
}