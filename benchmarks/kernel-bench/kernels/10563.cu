#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void cumprod_hybrid_kernel(
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ input,
    const int64_t dim_size,
    const int64_t stride,
    const int64_t total_batches) {
    
    extern __shared__ scalar_t shared_products[];
    
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x;
         idx < total_batches;
         idx += blockDim.x * gridDim.x) {
        
        const int batch_idx = idx / stride;
        const int in_idx = idx % stride;
        const int64_t base_idx = batch_idx * (dim_size * stride) + in_idx;
        
        scalar_t product = 1;
        int i = 0;
        
        #pragma unroll 8
        for (; i + 7 < dim_size && i < 32; i += 8) {
            const int64_t idx0 = base_idx + i * stride;
            product *= input[idx0];
            output[idx0] = product;
            
            const int64_t idx1 = base_idx + (i + 1) * stride;
            product *= input[idx1];
            output[idx1] = product;
            
            const int64_t idx2 = base_idx + (i + 2) * stride;
            product *= input[idx2];
            output[idx2] = product;
            
            const int64_t idx3 = base_idx + (i + 3) * stride;
            product *= input[idx3];
            output[idx3] = product;
            
            const int64_t idx4 = base_idx + (i + 4) * stride;
            product *= input[idx4];
            output[idx4] = product;
            
            const int64_t idx5 = base_idx + (i + 5) * stride;
            product *= input[idx5];
            output[idx5] = product;
            
            const int64_t idx6 = base_idx + (i + 6) * stride;
            product *= input[idx6];
            output[idx6] = product;
            
            const int64_t idx7 = base_idx + (i + 7) * stride;
            product *= input[idx7];
            output[idx7] = product;
        }
        
        for (; i < dim_size; i += 4) {
            if (i + 4 <= dim_size) {
                const int64_t curr_idx = base_idx + i * stride;
                float4 vec_in = *reinterpret_cast<const float4*>(&input[curr_idx]);
                
                product *= vec_in.x;
                output[curr_idx] = product;
                
                if (i + 1 < dim_size) {
                    product *= vec_in.y;
                    output[curr_idx + stride] = product;
                }
                if (i + 2 < dim_size) {
                    product *= vec_in.z;
                    output[curr_idx + 2 * stride] = product;
                }
                if (i + 3 < dim_size) {
                    product *= vec_in.w;
                    output[curr_idx + 3 * stride] = product;
                }
            } else {
                for (; i < dim_size; i++) {
                    const int64_t curr_idx = base_idx + i * stride;
                    product *= input[curr_idx];
                    output[curr_idx] = product;
                }
            }
        }
    }
}

torch::Tensor cumprod_cuda_hybrid_forward(torch::Tensor input, int64_t dim) {
    auto output = torch::empty_like(input);
    
    auto sizes = input.sizes();
    auto strides = input.strides();
    
    const int64_t dim_size = sizes[dim];
    const int64_t stride = strides[dim];
    const int64_t total_batches = input.numel() / dim_size;
    
    const int threads = 512;
    const int blocks = (total_batches + threads - 1) / threads;
    const int shared_mem_size = threads * sizeof(float);
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "cumprod_hybrid", ([&] {
        cumprod_hybrid_kernel<scalar_t><<<blocks, threads, shared_mem_size>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            dim_size,
            stride,
            total_batches
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &cumprod_cuda_hybrid_forward, "Hybrid optimized cumulative product forward (CUDA)");
}