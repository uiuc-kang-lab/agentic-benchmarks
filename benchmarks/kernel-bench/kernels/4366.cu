#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

__inline__ __device__ float warpReduceSum(float val) {
    #if __CUDA_ARCH__ >= 800
        return __reduce_add_sync(0xffffffff, val);
    #else
        for (int offset = warpSize/2; offset > 0; offset /= 2) {
            val += __shfl_down_sync(0xffffffff, val, offset);
        }
        return val;
    #endif
}

__inline__ __device__ float blockReduceSum(float val) {
    __shared__ float shared[32];
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    val = warpReduceSum(val);
    if (lane == 0) shared[wid] = val;
    __syncthreads();

    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0.0f;
    if (wid == 0) val = warpReduceSum(val);
    return val;
}

template<int BLOCK_SIZE = 256>
__global__ void instance_norm_kernel(
    const float* __restrict__ x,
    float* __restrict__ y,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    const int N,
    const int C,
    const int H,
    const int W,
    const float eps
) {
    const int HW = H * W;
    const int instance_idx = blockIdx.x;
    const int n = instance_idx / C;
    const int c = instance_idx % C;
    
    if (instance_idx >= N * C) return;

    const float w = weight ? weight[c] : 1.0f;
    const float b = bias ? bias[c] : 0.0f;

    const float* x_instance = x + (n * C + c) * HW;
    float* y_instance = y + (n * C + c) * HW;

    float sum = 0.0f;
    float sum_sq = 0.0f;
    
    const int vec_elements = HW / 4;
    const int vec_stride = BLOCK_SIZE;
    
    #pragma unroll 4
    for (int i = threadIdx.x; i < vec_elements; i += vec_stride) {
        float4 data = reinterpret_cast<const float4*>(x_instance)[i];
        sum += data.x + data.y + data.z + data.w;
        sum_sq += data.x * data.x + data.y * data.y + 
                  data.z * data.z + data.w * data.w;
    }
    
    const int rem_start = vec_elements * 4;
    for (int i = rem_start + threadIdx.x; i < HW; i += BLOCK_SIZE) {
        float val = x_instance[i];
        sum += val;
        sum_sq += val * val;
    }
    
    sum = blockReduceSum(sum);
    sum_sq = blockReduceSum(sum_sq);
    
    __shared__ float s_mean, s_inv_std;
    
    if (threadIdx.x == 0) {
        float mean = sum / HW;
        float var = fmaxf((sum_sq / HW) - (mean * mean), 0.0f);
        s_mean = mean;
        s_inv_std = rsqrtf(var + eps);
    }
    __syncthreads();
    
    const float mean = s_mean;
    const float inv_std = s_inv_std;
    
    #pragma unroll 4
    for (int i = threadIdx.x; i < vec_elements; i += vec_stride) {
        float4 data = reinterpret_cast<const float4*>(x_instance)[i];
        float4 result;
        result.x = ((data.x - mean) * inv_std * w) + b;
        result.y = ((data.y - mean) * inv_std * w) + b;
        result.z = ((data.z - mean) * inv_std * w) + b;
        result.w = ((data.w - mean) * inv_std * w) + b;
        reinterpret_cast<float4*>(y_instance)[i] = result;
    }
    
    for (int i = rem_start + threadIdx.x; i < HW; i += BLOCK_SIZE) {
        float val = x_instance[i];
        y_instance[i] = ((val - mean) * inv_std * w) + b;
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    double eps
) {
    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    if (weight.defined()) TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
    if (bias.defined()) TORCH_CHECK(bias.is_cuda(), "bias must be a CUDA tensor");
    
    auto sizes = x.sizes();
    TORCH_CHECK(sizes.size() == 4, "Input must be 4D");
    
    const int N = sizes[0], C = sizes[1], H = sizes[2], W = sizes[3];
    auto y = torch::empty_like(x);
    
    const int threads = 256;
    const dim3 blocks(N * C);
    
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    instance_norm_kernel<256><<<blocks, threads, 0, stream>>>(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        weight.defined() ? weight.data_ptr<float>() : nullptr,
        bias.defined() ? bias.data_ptr<float>() : nullptr,
        N, C, H, W,
        static_cast<float>(eps)
    );
    
    return y;
}