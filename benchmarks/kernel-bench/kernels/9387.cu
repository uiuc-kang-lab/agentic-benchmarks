#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Modular device function to compute convolution sum for a single output element
__device__ inline float compute_conv2d_sum(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    int b,
    int oc,
    int h_out,
    int w_out,
    int in_channels,
    int input_height,
    int input_width,
    int kernel_h,
    int kernel_w,
    int stride,
    int pad_h,
    int pad_w,
    int dilation_h,
    int dilation_w) {

    float sum = 0.0f;
    for (int ic = 0; ic < in_channels; ++ic) {
        for (int kh = 0; kh < kernel_h; ++kh) {
            for (int kw = 0; kw < kernel_w; ++kw) {
                int h_in = h_out * stride + kh * dilation_h - pad_h;
                int w_in = w_out * stride + kw * dilation_w - pad_w;
                if (h_in >= 0 && h_in < input_height && w_in >= 0 && w_in < input_width) {
                    int x_index = b * (in_channels * input_height * input_width)
                                + ic * (input_height * input_width)
                                + h_in * input_width + w_in;
                    int weight_index = oc * (in_channels * kernel_h * kernel_w)
                                     + ic * (kernel_h * kernel_w)
                                     + kh * kernel_w + kw;
                    sum += x[x_index] * weight[weight_index];
                }
            }
        }
    }
    return sum;
}

// Kernel that leverages the modular device function for computing convolution
__global__ void modular_conv2d_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int input_height,
    int input_width,
    int out_channels,
    int kernel_h,
    int kernel_w,
    int height_out,
    int width_out,
    int stride,
    int pad_h,
    int pad_w,
    int dilation_h,
    int dilation_w) {

    // Map threads to output width, height, and output channel
    int w_out = blockIdx.x * blockDim.x + threadIdx.x;
    int h_out = blockIdx.y * blockDim.y + threadIdx.y;
    int oc = blockIdx.z;

    if (w_out >= width_out || h_out >= height_out || oc >= out_channels) return;

    // Loop over batch dimension
    for (int b = 0; b < batch_size; ++b) {
        // Initialize with bias if provided
        float value = (bias != nullptr) ? bias[oc] : 0.0f;
        // Add convolution sum computed by the modular device function
        value += compute_conv2d_sum(
            x, weight, b, oc, h_out, w_out,
            in_channels, input_height, input_width,
            kernel_h, kernel_w, stride, pad_h, pad_w, dilation_h, dilation_w);
        
        int out_index = b * (out_channels * height_out * width_out)
                      + oc * (height_out * width_out)
                      + h_out * width_out + w_out;
        output[out_index] = value;
    }
}

// Forward function wrapping the CUDA kernel
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,  // optional bias
    int stride,
    std::tuple<int, int> padding,
    std::tuple<int, int> dilation) {

    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");

    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        TORCH_CHECK(bias->is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        bias_ptr = bias->data_ptr<float>();
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int input_height = x.size(2);
    int input_width = x.size(3);

    int out_channels = weight.size(0);
    int kernel_h = weight.size(2);
    int kernel_w = weight.size(3);

    int pad_h = std::get<0>(padding);
    int pad_w = std::get<1>(padding);
    int dilation_h = std::get<0>(dilation);
    int dilation_w = std::get<1>(dilation);

    int height_out = (input_height + 2 * pad_h - dilation_h * (kernel_h - 1) - 1) / stride + 1;
    int width_out = (input_width + 2 * pad_w - dilation_w * (kernel_w - 1) - 1) / stride + 1;

    auto output = torch::empty({batch_size, out_channels, height_out, width_out}, x.options());

    // Configure grid and block dimensions
    dim3 threads(16, 16);
    dim3 blocks((width_out + threads.x - 1) / threads.x,
                (height_out + threads.y - 1) / threads.y,
                out_channels);

    modular_conv2d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        input_height,
        input_width,
        out_channels,
        kernel_h,
        kernel_w,
        height_out,
        width_out,
        stride,
        pad_h,
        pad_w,
        dilation_h,
        dilation_w
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Modular Conv2D forward (CUDA) with device function refactor");
}
