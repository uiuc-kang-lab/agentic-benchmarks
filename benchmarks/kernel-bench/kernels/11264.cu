#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void smooth_l1_loss_kernel(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    int n_elements
) {
    const int elements_per_thread = 8;  // Increased from 4 to 8
    const int tid = threadIdx.x;
    const int block_offset = blockIdx.x * blockDim.x * elements_per_thread;
    const int idx_start = block_offset + tid * elements_per_thread;
    float thread_sum = 0.0f;

    #pragma unroll
    for (int j = 0; j < elements_per_thread; j++) {
        int idx = idx_start + j;
        if (idx < n_elements) {
            float diff = predictions[idx] - targets[idx];
            float abs_diff = fabsf(diff);
            thread_sum += (abs_diff < 1.0f) ? (0.5f * diff * diff) : (abs_diff - 0.5f);
        }
    }

    // Warp-level reduction using warp shuffle instructions
    for (int offset = 16; offset > 0; offset /= 2) {
        thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
    }

    // Each warp's leader writes its sum to shared memory (4 warps per 128 threads)
    __shared__ __align__(32) float warp_sums[4];
    int warp_id = tid / 32;
    if ((tid & 31) == 0) {
        warp_sums[warp_id] = thread_sum;
    }
    __syncthreads();

    // Thread 0 aggregates the sums from each warp and performs an atomic add
    if (tid == 0) {
        float block_sum = warp_sums[0] + warp_sums[1] + warp_sums[2] + warp_sums[3];
        atomicAdd(output, block_sum / n_elements);
    }
}

torch::Tensor smooth_l1_loss_cuda(
    torch::Tensor predictions,
    torch::Tensor targets
) {
    TORCH_CHECK(
        predictions.sizes() == targets.sizes(),
        "Input tensors must have the same shape"
    );
    TORCH_CHECK(
        predictions.is_contiguous() && targets.is_contiguous(),
        "Input tensors must be contiguous"
    );
    TORCH_CHECK(
        predictions.device().is_cuda() && targets.device().is_cuda(),
        "Inputs must be CUDA tensors"
    );

    int n = predictions.numel();
    auto output = torch::zeros({1}, predictions.options());

    const int block_size = 128;  // Optimized block size
    const int elements_per_thread = 8;
    const int grid_size = (n + block_size * elements_per_thread - 1) / (block_size * elements_per_thread);

    smooth_l1_loss_kernel<<<grid_size, block_size>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &smooth_l1_loss_cuda, "Smooth L1 Loss (CUDA)");
}