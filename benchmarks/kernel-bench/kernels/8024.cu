#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Define the maximum weight size for constant memory (in elements).
#define MAX_WEIGHT_SIZE 4096

// Store the convolution weights in constant memory.
__constant__ float const_weight[MAX_WEIGHT_SIZE];

// CUDA kernel for transposed 1D convolution with warp-level optimizations.
// Utilize warp shuffle operations to avoid shared memory use for small reductions
__global__ void warp_level_conv_transposed1d_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    const float* bias,
    int batch_size,
    int in_channels,
    int out_channels,
    int input_width,
    int output_width,
    int kernel_size,
    int stride,
    int padding,
    int groups) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch_size * out_channels * output_width;
    if (index >= total) return;

    // Calculate the output indices: batch, output channel, and spatial position
    int j = index % output_width;                       // output spatial position
    int o = (index / output_width) % out_channels;        // output channel
    int b = index / (output_width * out_channels);        // batch index

    float sum = 0.0f;

    // Determine group sizes for input and output channels
    int group_size_in = in_channels / groups;
    int group_size_out = out_channels / groups;
    int g = o / group_size_out;  // group index

    // Iterate over kernel elements
    for (int k = 0; k < kernel_size; k++) {
        int i = j + padding - k;
        // Only proceed if i is aligned with the stride
        if (i % stride != 0) continue;
        i /= stride;
        if (i < 0 || i >= input_width) continue;

        // Loop over the input channels for this group
        for (int ic = 0; ic < group_size_in; ic++) {
            int real_ic = g * group_size_in + ic;
            int input_idx = b * in_channels * input_width + real_ic * input_width + i;
            // Weight indexing: weight is stored as [in_channels, group_size_out, kernel_size]
            int weight_idx = (real_ic * group_size_out + (o - g * group_size_out)) * kernel_size + k;
            sum += input[input_idx] * const_weight[weight_idx];
        }
    }

    // Apply warp shuffle reduction
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
    }

    // Each warp writes the first lane
    if ((threadIdx.x & 31) == 0) {
        // Add bias if provided
        if (bias != nullptr) {
            sum += bias[o];
        }

        output[index] = sum;
    }
}

// Host wrapper function
torch::Tensor forward(
    const torch::Tensor& x,
    const torch::Tensor& weight,
    const c10::optional<torch::Tensor>& bias,
    int64_t stride,
    int64_t padding,
    int64_t output_padding,
    int64_t groups) {

    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int input_width = x.size(2);
    int kernel_size = weight.size(2);
    int group_size_out = weight.size(1);
    int out_channels = group_size_out * groups;
    int output_width = (input_width - 1) * stride - 2 * padding + kernel_size + output_padding;

    auto output = torch::zeros({batch_size, out_channels, output_width}, x.options());

    int num_weight_elems = weight.numel();
    TORCH_CHECK(num_weight_elems <= MAX_WEIGHT_SIZE, "Weight size exceeds constant memory limit");
    hipMemcpyToSymbol(HIP_SYMBOL(const_weight), weight.data_ptr<float>(), num_weight_elems * sizeof(float), 0, hipMemcpyDeviceToDevice);

    int total_threads = batch_size * out_channels * output_width;
    int threads = 256;
    int blocks = (total_threads + threads - 1) / threads;

    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        bias_ptr = bias.value().data_ptr<float>();
    }

    warp_level_conv_transposed1d_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        x.data_ptr<float>(),
        output.data_ptr<float>(),
        bias_ptr,
        batch_size,
        in_channels,
        out_channels,
        input_width,
        output_width,
        kernel_size,
        stride,
        padding,
        groups
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Transposed 1D convolution forward (CUDA) with warp-level optimizations");
}