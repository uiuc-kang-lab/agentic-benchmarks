#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32

// CUDA kernel for matrix multiplication
__global__ void matmul_kernel(const float* __restrict__ A,
                              const float* __restrict__ B,
                              float* __restrict__ C,
                              int N) {
    __shared__ float s_A[TILE_SIZE][TILE_SIZE];
    __shared__ float s_B[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;
    
    float value = 0;

    for (int i = 0; i < (N + TILE_SIZE - 1) / TILE_SIZE; ++i) {
        if (row < N && i * TILE_SIZE + tx < N)
            s_A[ty][tx] = A[row * N + i * TILE_SIZE + tx];
        else
            s_A[ty][tx] = 0.0f;

        if (col < N && i * TILE_SIZE + ty < N)
            s_B[ty][tx] = B[(i * TILE_SIZE + ty) * N + col];
        else
            s_B[ty][tx] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k)
            value += s_A[ty][k] * s_B[k][tx];

        __syncthreads();
    }

    if (row < N && col < N)
        C[row * N + col] = value;
}

// C++ interface
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    // Check that A and B are float tensors, 2D, square, on CUDA
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be of same size");

    int N = A.size(0);

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, A.get_device());
    auto C = torch::zeros({N, N}, options);

    // Launch the CUDA kernel
    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    matmul_kernel<<<blocks, threads>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), N);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix Multiplication (CUDA)");
}