#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256  // Experiment with block sizes: 32, 64, 128, 256, 512
#endif

__global__ void conv1d_kernel_shared(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int B,
    int in_channels,
    int in_size,
    int out_channels,
    int kernel_size,
    int out_size,
    int stride,
    int dilation
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = B * out_channels * out_size;
    if (idx >= total_elements) return;

    // Calculate indices
    int o = idx % out_size;
    idx /= out_size;
    int oc = idx % out_channels;
    int b = idx / out_channels;

    float sum = 0.0f;
    int base_input_idx = o * stride;
    int b_offset = b * (in_channels * in_size);
    int oc_offset = oc * (in_channels * kernel_size);

    // Loop over input channels and kernel
    for (int ic = 0; ic < in_channels; ic++) {
        int x_offset = b_offset + ic * in_size;
        int w_offset = oc_offset + ic * kernel_size;
        for (int k = 0; k < kernel_size; k++) {
            int input_pos = base_input_idx + k * dilation;
            if (input_pos < in_size) {
                sum += x[x_offset + input_pos] * weight[w_offset + k];
            }
        }
    }

    if (bias != nullptr) {
        sum += bias[oc];
    }

    int out_idx = b * (out_channels * out_size) + oc * out_size + o;
    output[out_idx] = sum;
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int dilation
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(weight.dim() == 3, "weight must be 3D");
    TORCH_CHECK(weight.size(1) == x.size(1), "Input channels mismatch");

    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        TORCH_CHECK(bias->dim() == 1, "bias must be 1D");
        TORCH_CHECK(bias->size(0) == weight.size(0), "Bias size mismatch");
    }

    int B = x.size(0);
    int in_channels = x.size(1);
    int in_size = x.size(2);
    int out_channels = weight.size(0);
    int kernel_size = weight.size(2);

    int out_size = (in_size - dilation * (kernel_size - 1) - 1) / stride + 1;
    TORCH_CHECK(out_size > 0, "Invalid output size");

    auto output = torch::empty({B, out_channels, out_size}, x.options());
    if (output.numel() == 0) return output;

    const float* x_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias ? bias->data_ptr<float>() : nullptr;
    float* output_ptr = output.data_ptr<float>();

    int total_elements = B * out_channels * out_size;
    int blocks = (total_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    conv1d_kernel<<<blocks, BLOCK_SIZE>>>(
        x_ptr, weight_ptr, bias_ptr, output_ptr,
        B, in_channels, in_size,
        out_channels, kernel_size, out_size,
        stride, dilation
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel launch error: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "1D convolution forward (CUDA) with optimal block size tuning");
}
