#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

// This kernel uses a 2D thread block for spatial dimensions and a 3D grid where the z-dimension covers the combined batch and channel indices.
// This ensures that threads in a warp (which vary along threadIdx.x) access consecutive output elements in memory, improving global memory coalescing.

template <typename scalar_t>
__global__ void depthwiseConv2DKernelCoalesced(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ w,
    const scalar_t* __restrict__ b,
    scalar_t* __restrict__ out,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int kernel_size,
    const int out_height,
    const int out_width,
    const int stride,
    const int padding) {

    // Use blockIdx.z to represent the combined (n, c) index.
    int bc = blockIdx.z;
    int c = bc % in_channels;
    int n = bc / in_channels;

    // Compute output spatial coordinates using 2D block indexing.
    int h_out = blockIdx.y * blockDim.y + threadIdx.y;
    int w_out = blockIdx.x * blockDim.x + threadIdx.x;

    if (h_out < out_height && w_out < out_width) {
        scalar_t value = 0;
        // Loop over the kernel window and accumulate.
        for (int kh = 0; kh < kernel_size; ++kh) {
            for (int kw = 0; kw < kernel_size; ++kw) {
                int h_in = h_out * stride - padding + kh;
                int w_in = w_out * stride - padding + kw;
                if (h_in >= 0 && h_in < in_height && w_in >= 0 && w_in < in_width) {
                    // Compute index for input x which is in (batch, channel, height, width) layout.
                    int x_index = ((n * in_channels + c) * in_height + h_in) * in_width + w_in;
                    // Weight layout: (in_channels, 1, kernel_size, kernel_size).
                    int w_index = (c * kernel_size + kh) * kernel_size + kw;
                    value += x[x_index] * w[w_index];
                }
            }
        }
        // Add bias (one per channel).
        value += b[c];

        // Write the result to the output tensor (layout: batch, channel, out_height, out_width).
        int out_index = ((n * in_channels + c) * out_height + h_out) * out_width + w_out;
        out[out_index] = value;
    }
}

// The forward implementation that sets up the 2D grid/block configuration for better coalesced memory access.

torch::Tensor forward_impl(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int groups) {
    
    // Depthwise convolution: groups should equal to in_channels.
    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);

    const int kernel_size = weight.size(2);  // weight shape: (in_channels, 1, kernel_size, kernel_size)
    const int out_height = (in_height + 2 * padding - kernel_size) / stride + 1;
    const int out_width  = (in_width  + 2 * padding - kernel_size) / stride + 1;

    auto out = torch::empty({batch_size, in_channels, out_height, out_width}, x.options());

    // Use a 2D thread block so that threads in a warp (varying in x) access contiguous output memory.
    const dim3 threads(16, 16);
    const dim3 blocks(
        (out_width + threads.x - 1) / threads.x,
        (out_height + threads.y - 1) / threads.y,
        batch_size * in_channels);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "depthwise_conv2d_forward_coalesced", ([&] {
        depthwiseConv2DKernelCoalesced<scalar_t><<<blocks, threads>>>(
            x.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            bias.data_ptr<scalar_t>(),
            out.data_ptr<scalar_t>(),
            batch_size, in_channels, in_height, in_width,
            kernel_size, out_height, out_width,
            stride, padding
        );
    }));

    return out;
}

// Wrap the forward implementation to handle optional bias.

torch::Tensor forward_wrap(
    torch::Tensor x,
    torch::Tensor weight,
    pybind11::object bias_obj,
    int stride,
    int padding,
    int groups) {
    torch::Tensor bias;
    if (bias_obj.is_none()) {
        bias = torch::zeros({x.size(1)}, x.options());
    } else {
        bias = bias_obj.cast<torch::Tensor>();
    }
    
    return forward_impl(x, weight, bias, stride, padding, groups);
}

namespace py = pybind11;

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        &forward_wrap,
        "Depthwise conv2d forward with memory coalescing via 2D blocking",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = 1,
        py::arg("padding") = 0,
        py::arg("groups") = 1
    );
}
