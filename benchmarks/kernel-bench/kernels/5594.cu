#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t, int KERNEL_SIZE>
__global__ void max_pool2d_tiled_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int stride,
    const int padding,
    const int dilation
) {
    // 2D block organization for better thread utilization
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int bz = blockIdx.z;

    // Each block handles a 8x8 tile of the output
    const int TILE_SIZE = 8;
    const int oh = by * TILE_SIZE + ty;
    const int ow = bx * TILE_SIZE + tx;
    
    // Calculate batch and channel indices
    const int b = bz / channels;
    const int c = bz % channels;

    if (oh >= output_height || ow >= output_width) return;

    const int input_batch_offset = b * channels * input_height * input_width;
    const int input_channel_offset = c * input_height * input_width;

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    
    if constexpr (KERNEL_SIZE == 2) {
        #pragma unroll
        for (int kh = 0; kh < 2; kh++) {
            const int ih = oh * stride - padding + kh * dilation;
            if (ih >= 0 && ih < input_height) {
                const int row_offset = input_batch_offset + input_channel_offset + ih * input_width;
                #pragma unroll
                for (int kw = 0; kw < 2; kw++) {
                    const int iw = ow * stride - padding + kw * dilation;
                    if (iw >= 0 && iw < input_width) {
                        max_val = max(max_val, __ldg(&input[row_offset + iw]));
                    }
                }
            }
        }
    }
    else if constexpr (KERNEL_SIZE == 3) {
        #pragma unroll
        for (int kh = 0; kh < 3; kh++) {
            const int ih = oh * stride - padding + kh * dilation;
            if (ih >= 0 && ih < input_height) {
                const int row_offset = input_batch_offset + input_channel_offset + ih * input_width;
                #pragma unroll
                for (int kw = 0; kw < 3; kw++) {
                    const int iw = ow * stride - padding + kw * dilation;
                    if (iw >= 0 && iw < input_width) {
                        max_val = max(max_val, __ldg(&input[row_offset + iw]));
                    }
                }
            }
        }
    }
    else {
        for (int kh = 0; kh < KERNEL_SIZE; kh++) {
            const int ih = oh * stride - padding + kh * dilation;
            if (ih >= 0 && ih < input_height) {
                const int row_offset = input_batch_offset + input_channel_offset + ih * input_width;
                #pragma unroll 4
                for (int kw = 0; kw < KERNEL_SIZE; kw++) {
                    const int iw = ow * stride - padding + kw * dilation;
                    if (iw >= 0 && iw < input_width) {
                        max_val = max(max_val, __ldg(&input[row_offset + iw]));
                    }
                }
            }
        }
    }

    // Write output with coalesced access pattern
    const int output_idx = (b * channels * output_height * output_width) +
                          (c * output_height * output_width) +
                          (oh * output_width) + ow;
    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    // Use 8x8 thread blocks for better occupancy
    const dim3 threads(8, 8);
    const dim3 blocks(
        (output_width + threads.x - 1) / threads.x,
        (output_height + threads.y - 1) / threads.y,
        batch_size * channels
    );

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        if (kernel_size == 2) {
            max_pool2d_tiled_kernel<scalar_t, 2><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size, channels,
                input_height, input_width,
                output_height, output_width,
                stride, padding, dilation
            );
        }
        else if (kernel_size == 3) {
            max_pool2d_tiled_kernel<scalar_t, 3><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size, channels,
                input_height, input_width,
                output_height, output_width,
                stride, padding, dilation
            );
        }
        else {
            max_pool2d_tiled_kernel<scalar_t, -1><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size, channels,
                input_height, input_width,
                output_height, output_width,
                stride, padding, dilation
            );
        }
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}