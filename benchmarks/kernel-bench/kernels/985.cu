#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
#define ELEMENTS_PER_THREAD 4

__constant__ int const_dims[6];  // M, N, K, lda, ldb, ldc
__constant__ bool const_trans[2]; // transA, transB

__device__ float get_element(const float* __restrict__ matrix, int row, int col, int ld, bool transpose) {
    return transpose ? matrix[col * ld + row] : matrix[row * ld + col];
}

__global__ void matmul_kernel_warp(const float* __restrict__ A,
                                    const float* __restrict__ B,
                                    float* __restrict__ C) {
    const int M = const_dims[0];
    const int N = const_dims[1];
    const int K = const_dims[2];
    const int lda = const_dims[3];
    const int ldb = const_dims[4];
    const int ldc = const_dims[5];
    const bool transA = const_trans[0];
    const bool transB = const_trans[1];

    int block_row = blockIdx.y * (BLOCK_SIZE * ELEMENTS_PER_THREAD);
    int block_col = blockIdx.x * BLOCK_SIZE;
    int thread_row = threadIdx.y;
    int thread_col = threadIdx.x;

    float C_values[ELEMENTS_PER_THREAD] = {0.0f};

    for (int t = 0; t < (K + BLOCK_SIZE - 1) / BLOCK_SIZE; ++t) {
        float A_values[ELEMENTS_PER_THREAD];
        float B_value = 0.0f;

        #pragma unroll
        for (int e = 0; e < ELEMENTS_PER_THREAD; ++e) {
            int row = block_row + e * BLOCK_SIZE + thread_row;
            if (row < M && t * BLOCK_SIZE + thread_col < K) {
                A_values[e] = get_element(A, row, t * BLOCK_SIZE + thread_col, lda, transA);
            } else {
                A_values[e] = 0.0f;
            }
        }

        if (t * BLOCK_SIZE + thread_row < K && block_col + thread_col < N) {
            B_value = get_element(B, t * BLOCK_SIZE + thread_row, block_col + thread_col, ldb, transB);
        }

        #pragma unroll
        for (int e = 0; e < ELEMENTS_PER_THREAD; ++e) {
            #pragma unroll
            for (int k = 0; k < BLOCK_SIZE; ++k) {
                float b_shuffled = __shfl_sync(0xFFFFFFFF, B_value, k);
                C_values[e] += A_values[e] * b_shuffled;
            }
        }
    }

    #pragma unroll
    for (int e = 0; e < ELEMENTS_PER_THREAD; ++e) {
        int row = block_row + e * BLOCK_SIZE + thread_row;
        int col = block_col + thread_col;
        if (row < M && col < N) {
            C[row * ldc + col] = C_values[e];
        }
    }
}

torch::Tensor matmul_cuda(torch::Tensor A, torch::Tensor B) {
    if (!A.is_cuda() || !B.is_cuda()) {
        throw std::invalid_argument("Input tensors must be on CUDA devices");
    }

    int dims[6];
    dims[0] = A.size(0);  // M
    dims[1] = B.size(1);  // N
    dims[2] = A.size(1);  // K
    dims[3] = A.stride(0); // lda
    dims[4] = B.stride(0); // ldb
    dims[5] = B.size(1);   // ldc

    bool trans[2] = {false, false};

    // Copy configuration to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(const_dims), dims, sizeof(dims));
    hipMemcpyToSymbol(HIP_SYMBOL(const_trans), trans, sizeof(trans));

    auto C = torch::empty({dims[0], dims[1]}, A.options());

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((dims[1] + BLOCK_SIZE - 1) / BLOCK_SIZE,
                 (dims[0] + (BLOCK_SIZE * ELEMENTS_PER_THREAD) - 1) / (BLOCK_SIZE * ELEMENTS_PER_THREAD));

    matmul_kernel_warp<<<gridDim, blockDim>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>());

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_cuda, "Matrix multiplication with warp-level primitives (CUDA)");
}
