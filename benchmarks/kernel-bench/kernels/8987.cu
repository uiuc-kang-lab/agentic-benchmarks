#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Optimized 1D convolution kernel that combines shared memory caching of weights and loop unrolling for the kernel loop.
// Each block computes the output for a single (batch, output channel) pair. The corresponding weight filter is loaded
// once into shared memory, then threads in the block cooperatively compute outputs along the width dimension.

__global__ void conv1d_kernel_optimized(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int in_channels,
    int in_size,
    int out_size,
    int kernel_size,
    int stride,
    int dilation
) {
    // Identify batch index and output channel from grid configuration
    int b = blockIdx.x;
    int oc = blockIdx.y;

    // Compute total number of weights per filter
    int filter_size = in_channels * kernel_size;

    // Allocate shared memory for the weight filter for this output channel
    extern __shared__ float sweight[];  // size: in_channels * kernel_size

    // Load weight filter into shared memory
    for (int i = threadIdx.x; i < filter_size; i += blockDim.x) {
        sweight[i] = weight[oc * filter_size + i];
    }
    __syncthreads();

    // Load bias value for this output channel if available
    float bias_val = (bias != nullptr) ? bias[oc] : 0.0f;

    // Each thread computes for one or more output positions along the width dimension
    for (int o = threadIdx.x; o < out_size; o += blockDim.x) {
        float sum = 0.0f;
        // Loop over each input channel
        for (int ic = 0; ic < in_channels; ++ic) {
            // Unroll the inner loop over kernel positions to reduce loop overhead
            #pragma unroll
            for (int k = 0; k < kernel_size; ++k) {
                int input_pos = o * stride + k * dilation;
                if (input_pos < in_size) {
                    int x_index = b * (in_channels * in_size) + ic * in_size + input_pos;
                    int w_index = ic * kernel_size + k;  // Index in shared memory
                    sum += x[x_index] * sweight[w_index];
                }
            }
        }
        sum += bias_val;
        // Compute global output index: (B, out_channels, out_size)
        int out_index = b * (gridDim.y * out_size) + oc * out_size + o;
        output[out_index] = sum;
    }
}

// Forward function routed via pybind11
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int dilation
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");
    TORCH_CHECK(x.dim() == 3, "x must be 3D");
    TORCH_CHECK(weight.dim() == 3, "weight must be 3D");
    TORCH_CHECK(weight.size(1) == x.size(1), "Input channels mismatch");

    if (bias.has_value()) {
        TORCH_CHECK(bias->device().is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        TORCH_CHECK(bias->dim() == 1, "bias must be 1D");
        TORCH_CHECK(bias->size(0) == weight.size(0), "Bias size mismatch");
    }

    int B = x.size(0);
    int in_channels = x.size(1);
    int in_size = x.size(2);
    int out_channels = weight.size(0);
    int kernel_size = weight.size(2);

    int out_size = (in_size - dilation * (kernel_size - 1) - 1) / stride + 1;
    TORCH_CHECK(out_size > 0, "Invalid output size");

    auto output = torch::empty({B, out_channels, out_size}, x.options());
    if (output.numel() == 0) return output;

    const float* x_data = x.data_ptr<float>();
    const float* weight_data = weight.data_ptr<float>();
    const float* bias_data = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float* output_data = output.data_ptr<float>();

    // Configure a 2D grid: one block per (batch, output channel)
    dim3 blocks(B, out_channels);
    int threads = 256;
    // Shared memory size for storing one weight filter
    int shared_mem_size = in_channels * kernel_size * sizeof(float);

    conv1d_kernel_optimized<<<blocks, threads, shared_mem_size>>>(
        x_data,
        weight_data,
        bias_data,
        output_data,
        in_channels,
        in_size,
        out_size,
        kernel_size,
        stride,
        dilation
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel launch error: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized 1D convolution forward (CUDA) using shared memory and loop unrolling");
}
