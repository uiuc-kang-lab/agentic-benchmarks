#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// New kernel using grid-stride loop for improved load balancing across threads

template <typename scalar_t>
__global__ void conv_transpose2d_kernel_gridstride(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int out_channels,
    const int kernel_h,
    const int kernel_w,
    const int stride,
    const int padding,
    const int output_padding,
    const int groups,
    const int dilation,
    const int out_height,
    const int out_width
) {
    const int total_elements = batch_size * out_channels * out_height * out_width;
    
    // Grid-stride loop to distribute workload evenly
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < total_elements; idx += blockDim.x * gridDim.x) {
        int n = idx;
        const int ow = n % out_width;
        n /= out_width;
        const int oh = n % out_height;
        n /= out_height;
        const int oc = n % out_channels;
        n /= out_channels;
        const int b = n;
        
        // Determine group and channel offset
        const int out_channels_per_group = out_channels / groups;
        const int g = oc / out_channels_per_group;
        const int oc_group = oc % out_channels_per_group;
        const int in_channels_per_group = in_channels / groups;
        const int ic_start = g * in_channels_per_group;
        
        // Initialize accumulation with bias if provided
        scalar_t val = (bias != nullptr) ? bias[oc] : static_cast<scalar_t>(0);
        
        // Loop over kernel dimensions
        for (int kh = 0; kh < kernel_h; ++kh) {
            for (int kw = 0; kw < kernel_w; ++kw) {
                int h_offset = oh - kh * dilation + padding;
                if (h_offset < 0 || h_offset % stride != 0) continue;
                int h_in = h_offset / stride;
                if (h_in < 0 || h_in >= in_height) continue;
                
                int w_offset = ow - kw * dilation + padding;
                if (w_offset < 0 || w_offset % stride != 0) continue;
                int w_in = w_offset / stride;
                if (w_in < 0 || w_in >= in_width) continue;
                
                // Accumulate over the corresponding input channels
                for (int ic = 0; ic < in_channels_per_group; ++ic) {
                    const int input_idx = b * in_channels * in_height * in_width
                                            + (ic_start + ic) * in_height * in_width
                                            + h_in * in_width
                                            + w_in;
                    const scalar_t x_val = input[input_idx];
                    
                    const int weight_idx = (ic_start + ic) * (out_channels_per_group * kernel_h * kernel_w)
                                             + oc_group * (kernel_h * kernel_w)
                                             + kh * kernel_w + kw;
                    const scalar_t w_val = weight[weight_idx];
                    
                    val += x_val * w_val;
                }
            }
        }
        output[idx] = val;
    }
}

// Forward function wrapping the CUDA kernel launch

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int output_padding,
    int groups,
    int dilation = 1
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(x.dim() == 4, "Input must be 4D");
    TORCH_CHECK(weight.dim() == 4, "Weight must be 4D");

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);

    // Compute the number of output channels based on groups
    const int out_channels = weight.size(1) * groups;  // weight shape: [in_channels, out_channels/groups, kH, kW]
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);

    if (bias.has_value() && bias->defined()) {
        TORCH_CHECK(bias->numel() == out_channels, "Bias must have out_channels elements");
        TORCH_CHECK(bias->device().is_cuda(), "Bias must be a CUDA tensor");
    }

    // Calculate output spatial dimensions
    const int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_h - 1) + output_padding + 1;
    const int out_width = (in_width - 1) * stride - 2 * padding + dilation * (kernel_w - 1) + output_padding + 1;

    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, x.options());

    const int total_elements = output.numel();
    const int threads = 1024;
    const int blocks = (total_elements + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "conv_transpose2d_cuda", ([&] {
        conv_transpose2d_kernel_gridstride<scalar_t><<<blocks, threads>>>(
            x.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            (bias.has_value() && bias->defined()) ? bias->data_ptr<scalar_t>() : nullptr,
            output.data_ptr<scalar_t>(),
            batch_size,
            in_channels,
            in_height,
            in_width,
            out_channels,
            kernel_h,
            kernel_w,
            stride,
            padding,
            output_padding,
            groups,
            dilation,
            out_height,
            out_width
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Transposed 2D convolution with grid-stride loop (CUDA)",
          py::arg("x"), py::arg("weight"), py::arg("bias") = py::none(),
          py::arg("stride"), py::arg("padding"), py::arg("output_padding"),
          py::arg("groups"), py::arg("dilation") = 1);
}
