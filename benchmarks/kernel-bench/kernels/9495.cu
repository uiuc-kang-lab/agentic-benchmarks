#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// This kernel performs the 2D transposed convolution operation with manual loop unrolling via #pragma unroll.
// The inner loops (over kernel height and width) are unrolled to reduce loop overhead and improve performance.

__global__ void conv_transpose2d_forward_kernel_unrolled_manual(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

  // Decode the linear index into output tensor coordinates (b, o, out_h, out_w)
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch_size * out_channels * out_height * out_width;
  if (index >= total) return;

  int w_out = index % out_width;
  int temp = index / out_width;
  int h_out = temp % out_height;
  temp /= out_height;
  int o = temp % out_channels;
  int b = temp / out_channels;

  // Start with bias for the output channel
  float result = bias[o];

  // Manually unroll the loops for kernel height and width
  #pragma unroll
  for (int c = 0; c < in_channels; c++) {
    #pragma unroll
    for (int p = 0; p < kernel_size; p++) {
      int h_unscaled = h_out + padding - p * dilation;
      if (h_unscaled % stride != 0) continue;
      int h_in = h_unscaled / stride;
      if (h_in < 0 || h_in >= in_height) continue;
      
      #pragma unroll
      for (int q = 0; q < kernel_size; q++) {
        int w_unscaled = w_out + padding - q * dilation;
        if (w_unscaled % stride != 0) continue;
        int w_in = w_unscaled / stride;
        if (w_in < 0 || w_in >= in_width) continue;
        
        int input_idx = ((b * in_channels + c) * in_height + h_in) * in_width + w_in;
        int weight_idx = ((c * out_channels + o) * kernel_size + p) * kernel_size + q;
        result += input[input_idx] * weight[weight_idx];
      }
    }
  }

  int output_idx = ((b * out_channels + o) * out_height + h_out) * out_width + w_out;
  output[output_idx] = result;
}

// CUDA launcher function
torch::Tensor conv_transpose2d_forward_cuda_unrolled_manual(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {
  
  int batch_size = input.size(0);
  int in_channels = input.size(1);
  int in_height = input.size(2);
  int in_width = input.size(3);
  
  int out_channels = weight.size(1);
  int kernel_size = weight.size(2);  // assume square kernel
  
  int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  int out_width  = (in_width - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  
  auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());
  
  int total_threads = batch_size * out_channels * out_height * out_width;
  int threads = 1024;
  int blocks = (total_threads + threads - 1) / threads;
  
  conv_transpose2d_forward_kernel_unrolled_manual<<<blocks, threads>>>(
      input.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      output.data_ptr<float>(),
      batch_size,
      in_channels,
      out_channels,
      in_height,
      in_width,
      kernel_size,
      out_height,
      out_width,
      stride,
      padding,
      dilation);
      
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in conv_transpose2d_forward_kernel_unrolled_manual: %s\n", hipGetErrorString(err));
  }
  
  return output;
}

// Wrapper function to handle bias being potentially None
torch::Tensor conv_transpose2d_forward_wrapper_unrolled_manual(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,
    int stride,
    int padding,
    int dilation) {
  
  int out_channels = weight.size(1);
  torch::Tensor bias;
  if (bias_obj.is(pybind11::none())) {
    bias = torch::zeros({out_channels}, weight.options());
  } else {
    bias = bias_obj.cast<torch::Tensor>();
  }
  
  return conv_transpose2d_forward_cuda_unrolled_manual(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &conv_transpose2d_forward_wrapper_unrolled_manual,
        "ConvTranspose2d forward with manually unrolled loops (CUDA)",
        pybind11::arg("input"),
        pybind11::arg("weight"),
        pybind11::arg("bias"),
        pybind11::arg("stride"),
        pybind11::arg("padding"),
        pybind11::arg("dilation"));
}
