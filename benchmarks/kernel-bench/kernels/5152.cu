#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ATen/AccumulateType.h>
#include <math.h>

// Optimized CUDA kernel for LayerNorm forward using warp-level primitives to reduce warp divergence.

template <typename scalar_t>
__global__ void optimized_layernorm_forward_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    const float eps,
    scalar_t* __restrict__ output,
    const int normalized_size) {

  // Each block processes one instance along the outer dimension.
  int instance_idx = blockIdx.x;
  int tid = threadIdx.x;

  const scalar_t* in_ptr = input + instance_idx * normalized_size;
  scalar_t* out_ptr = output + instance_idx * normalized_size;

  using accscalar_t = at::acc_type<scalar_t, true>;

  // Static shared memory for broadcasting the computed mean and inverse std
  __shared__ accscalar_t shared_mean;
  __shared__ accscalar_t shared_inv_std;

  // Each thread computes partial sum and sum-of-squares over its strided load
  accscalar_t local_sum = 0;
  accscalar_t local_sum_sq = 0;
  for (int i = tid; i < normalized_size; i += blockDim.x) {
    accscalar_t val = static_cast<accscalar_t>(in_ptr[i]);
    local_sum += val;
    local_sum_sq += val * val;
  }

  // Use warp-level shuffle to reduce within each warp, minimizing divergent branching.
  unsigned int mask = 0xFFFFFFFF;
  int lane = tid & 31;
  for (int offset = 16; offset > 0; offset /= 2) {
    local_sum    += __shfl_down_sync(mask, local_sum, offset);
    local_sum_sq += __shfl_down_sync(mask, local_sum_sq, offset);
  }

  // Compute warp id and number of warps in this block
  int warp_id = tid >> 5;  // equivalent to tid / 32
  int numWarps = (blockDim.x + 31) / 32;

  // Use dynamic shared memory to store partial sums from each warp
  extern __shared__ char smem[];
  accscalar_t* s_sum    = reinterpret_cast<accscalar_t*>(smem);
  accscalar_t* s_sum_sq = s_sum + numWarps;

  // The first lane of each warp writes its result to shared memory
  if (lane == 0) {
    s_sum[warp_id] = local_sum;
    s_sum_sq[warp_id] = local_sum_sq;
  }
  __syncthreads();

  // Final reduction from warp-level partial sums performed by thread 0
  if (tid == 0) {
    accscalar_t total_sum = 0;
    accscalar_t total_sum_sq = 0;
    for (int i = 0; i < numWarps; i++) {
      total_sum    += s_sum[i];
      total_sum_sq += s_sum_sq[i];
    }
    accscalar_t mean = total_sum / static_cast<accscalar_t>(normalized_size);
    accscalar_t var = total_sum_sq / static_cast<accscalar_t>(normalized_size) - mean * mean;
    shared_mean = mean;
    shared_inv_std = static_cast<accscalar_t>(1) / __fsqrt_rn(var + static_cast<accscalar_t>(eps));
  }
  __syncthreads();

  // Each thread normalizes its chunk and applies the affine transformation
  for (int i = tid; i < normalized_size; i += blockDim.x) {
    accscalar_t val = static_cast<accscalar_t>(in_ptr[i]);
    accscalar_t norm_val = (val - shared_mean) * shared_inv_std;
    out_ptr[i] = static_cast<scalar_t>(norm_val * static_cast<accscalar_t>(weight[i]) + 
                                        static_cast<accscalar_t>(bias[i]));
  }
}

// Host function that launches the optimized kernel

torch::Tensor optimized_layernorm_forward(torch::Tensor x, torch::Tensor weight, torch::Tensor bias, double eps = 1e-5) {
  // Create output tensor with the same shape as x
  auto output = torch::empty_like(x);

  int normalized_size = weight.numel();
  int outer_size = x.numel() / normalized_size;

  // Choose the number of threads (up to 1024 per block)
  int threads = (normalized_size < 1024) ? normalized_size : 1024;
  int blocks = outer_size;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "optimized_layernorm_forward_cuda", ([&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    int warpCount = (threads + 31) / 32;
    int shared_size = warpCount * 2 * sizeof(accscalar_t);
    optimized_layernorm_forward_kernel<scalar_t><<<blocks, threads, shared_size>>>(
        x.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(),
        bias.data_ptr<scalar_t>(),
        static_cast<float>(eps),
        output.data_ptr<scalar_t>(),
        normalized_size);
  }));

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &optimized_layernorm_forward, "Optimized LayerNorm forward (CUDA)",
        py::arg("x"), py::arg("weight"), py::arg("bias"), py::arg("eps") = 1e-5);
}
