#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

// Optimized CUDA kernel for matrix multiplication using shared memory and warp-level reductions

template <typename scalar_t>
__global__ void matmul_cuda_kernel(const scalar_t* __restrict__ A, const scalar_t* __restrict__ B,
                                   scalar_t* __restrict__ C, int M, int K, int N) {
    __shared__ scalar_t sA[TILE_WIDTH][TILE_WIDTH];
    __shared__ scalar_t sB[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y; // Index in M dimension
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x; // Index in N dimension

    scalar_t value = 0;
    int num_tiles = (K + TILE_WIDTH - 1) / TILE_WIDTH;

    for (int t = 0; t < num_tiles; ++t) {
        int tiledA_col = t * TILE_WIDTH + threadIdx.x;
        int tiledB_row = t * TILE_WIDTH + threadIdx.y;

        if (row < M && tiledA_col < K)
            sA[threadIdx.y][threadIdx.x] = A[row * K + tiledA_col];
        else
            sA[threadIdx.y][threadIdx.x] = 0;

        if (col < N && tiledB_row < K)
            sB[threadIdx.y][threadIdx.x] = B[tiledB_row * N + col];
        else
            sB[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();

        #pragma unroll
        for (int i = 0; i < TILE_WIDTH; ++i) {
            value += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Warp-level reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        value += __shfl_down_sync(0xFFFFFFFF, value, offset);
    }

    // Write to output
    if (row < M && col < N && (threadIdx.x % warpSize) == 0) {
        C[row * N + col] = value;
    }
}

// Forward function

torch::Tensor module_fn(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input tensor B must be a CUDA tensor");

    int64_t M = A.size(0);
    int64_t K = A.size(1);
    int64_t N = B.size(1);
    TORCH_CHECK(K == B.size(0), "Inner dimensions of A and B must match");

    // Allocate output tensor
    auto C = torch::empty({M, N}, A.options());

    dim3 threads_per_block(TILE_WIDTH, TILE_WIDTH);
    dim3 num_blocks((N + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matmul_cuda_kernel", ([&] {
        matmul_cuda_kernel<scalar_t><<<num_blocks, threads_per_block>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, K, N);
    }));

    hipDeviceSynchronize();
    return C;
}

// Pybind11 module binding
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &module_fn, "Optimized matrix multiplication forward (CUDA, warp-level reduction)");
}
