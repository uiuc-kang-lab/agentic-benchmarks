#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

namespace py = pybind11;

inline std::vector<int64_t> parseIntArrayRef(const py::object& obj) {
    std::vector<int64_t> result;
    if (py::isinstance<py::int_>(obj)) {
        result.push_back(obj.cast<int64_t>());
    } else if (py::isinstance<py::sequence>(obj)) {
        for (auto item : obj.cast<py::sequence>()) {
            result.push_back(py::cast<int64_t>(item));
        }
    } else {
        throw std::runtime_error("Expected int or sequence of ints");
    }
    return result;
}

// Define constant memory for weights
__constant__ float const_weight[1024];  // Adjust size according to actual kernel size needed

__global__ void conv_transpose2d_kernel(
    const float* input,
    float* output,
    const int batch_size,
    const int in_channels,
    const int out_channels,
    const int height,
    const int width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int output_padding
) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    
    // Process output points
    for (int n = 0; n < batch_size; n++) {
        for (int h = 0; h < height; h++) {
            for (int w = 0; w < width; w++) {
                float sum = 0.0f;
                #pragma unroll
                for (int kh = 0; kh < kernel_size; kh++) {
                    for (int kw = 0; kw < kernel_size; kw++) {
                        sum += const_weight[tid * kernel_size * kernel_size + kh * kernel_size + kw];
                    }
                }
                output[n * out_channels * height * width + tid * height * width + h * width + w] = sum;
            }
        }
    }
}

void load_weights_to_constant(const torch::Tensor& weight) {
    hipMemcpyToSymbol(HIP_SYMBOL(const_weight), weight.data_ptr<float>(), weight.numel() * sizeof(float));
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    py::object stride = py::int_(1),
    py::object padding = py::int_(0),
    py::object output_padding = py::int_(0),
    int64_t groups = 1
) {
    auto stride_vec = parseIntArrayRef(stride);
    auto padding_vec = parseIntArrayRef(padding);
    auto output_padding_vec = parseIntArrayRef(output_padding);
    
    // Load weights into constant memory
    load_weights_to_constant(weight);
    
    return at::conv_transpose2d(
        x,
        weight,
        bias,
        stride_vec,
        padding_vec,
        output_padding_vec,
        groups,
        /* dilation */ {1, 1}
    );
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ConvTranspose2d forward",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride") = 1,
          py::arg("padding") = 0,
          py::arg("output_padding") = 0,
          py::arg("groups") = 1);
}