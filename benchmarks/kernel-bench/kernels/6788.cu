#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define BLOCK_SIZE 256

__global__ void prod_reduce_kernel(const float* __restrict__ input, 
                                 float* __restrict__ output, 
                                 const int stride, 
                                 const int num_elements) {
    int idx = blockIdx.x * blockDim.x * 50 + threadIdx.x;
    float product = 1.0f;

    // Loop over the entire workload, blocking by blockDim.x and thread count
    for (int i = idx; i < num_elements; i += blockDim.x * gridDim.x * 50) {
        float local_product = 1.0f;
        int local_offset = i;
        #pragma unroll
        for (int j = 0; j < 50 && (local_offset + j * stride) < num_elements; ++j) {
            local_product *= input[local_offset + j * stride];
        }
        product *= local_product;
    }

    if (idx < num_elements) {
        output[blockIdx.x] = product;
    }
}

torch::Tensor forward(torch::Tensor x, int dim) {
    CHECK_INPUT(x);

    auto sizes = x.sizes().vec();
    int dim_size = sizes[dim];
    sizes.erase(sizes.begin() + dim);
    torch::Tensor output = torch::empty(sizes, x.options());

    int num_elements = output.numel();
    int stride = x.stride(dim);

    const float* input_ptr = x.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();

    int blocks = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    prod_reduce_kernel<<<blocks, BLOCK_SIZE>>>(input_ptr, output_ptr, stride, num_elements);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Product reduction over a dimension (CUDA)");
}
