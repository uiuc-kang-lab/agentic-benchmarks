#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <type_traits>

// Branchless clamp function using CUDA intrinsics to minimize warp divergence
template <typename scalar_t>
__device__ inline scalar_t clamp_val(scalar_t x) {
  if constexpr (std::is_same<scalar_t, float>::value) {
    return fminf(fmaxf(x, 0.f), 1.f);
  } else {
    return fmin(fmax(x, static_cast<scalar_t>(0)), static_cast<scalar_t>(1));
  }
}

// CUDA kernel: computes HardSigmoid activation: y = clamp((x + 3) / 6, 0, 1) 
// using branchless intrinsics to reduce warp divergence
template <typename scalar_t>
__global__ void branchless_hardsigmoid_kernel(const scalar_t* __restrict__ input,
                                               scalar_t* __restrict__ output,
                                               size_t numel) {
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;
  for (size_t i = idx; i < numel; i += stride) {
    const scalar_t x = input[i];
    scalar_t y = (x + static_cast<scalar_t>(3)) / static_cast<scalar_t>(6);
    // Apply branchless clamp to maintain uniform control flow
    y = clamp_val(y);
    output[i] = y;
  }
}

// Forward function called from Python
torch::Tensor forward(torch::Tensor input) {
  TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");
  auto output = torch::empty_like(input);
  const size_t numel = input.numel();
  const int threads = 1024;
  const int blocks = (numel + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "branchless_hardsigmoid_cuda", ([&] {
    branchless_hardsigmoid_kernel<scalar_t><<<blocks, threads>>>(
        input.data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>(),
        numel);
  }));

  hipError_t err = hipGetLastError();
  TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "HardSigmoid activation forward (CUDA) with branchless clamping");
}
