#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

__global__ void conv2d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    float* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int out_channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int stride,
    const int padding) {

    const int out_h = (height + 2 * padding - kernel_h) / stride + 1;
    const int out_w = (width + 2 * padding - kernel_w) / stride + 1;

    const int w_out = blockIdx.x * blockDim.x + threadIdx.x;
    const int h_out = blockIdx.y * blockDim.y + threadIdx.y;
    const int oc = blockIdx.z;

    if (w_out < out_w && h_out < out_h && oc < out_channels) {
        for (int b = 0; b < batch_size; ++b) {
            float sum = 0.0f; __shared__ float shared_weight[16][16];
            for (int ic = 0; ic < in_channels; ++ic) {
                for (int kh = 0; kh < kernel_h; ++kh) {
                    for (int kw = 0; kw < kernel_w; ++kw) {
                        int h_in = h_out * stride + kh - padding;
                        int w_in = w_out * stride + kw - padding;
                        if (h_in >= 0 && h_in < height && w_in >= 0 && w_in < width) {
                            float input_val = __ldg(&input[((b * in_channels + ic) * height + h_in) * width + w_in]);
                            float weight_val = __ldg(&weight[((oc * in_channels + ic) * kernel_h + kh) * kernel_w + kw]);
                            sum += input_val * weight_val;
                        }
                    }
                }
            }
            output[((b * out_channels + oc) * out_h + h_out) * out_w + w_out] = sum;
        }
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    TORCH_CHECK(x.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "Weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "Input must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "Weight must be contiguous");

    if (dilation != 1 || groups != 1) {
        return torch::conv2d(x, weight, bias,
                             {stride, stride}, {padding, padding}, {dilation, dilation}, groups);
    }

    auto batch_size = x.size(0);
    auto in_channels = x.size(1);
    auto height = x.size(2);
    auto width = x.size(3);
    auto out_channels = weight.size(0);
    auto kernel_h = weight.size(2);
    auto kernel_w = weight.size(3);

    const int out_h = (height + 2 * padding - kernel_h) / stride + 1;
    const int out_w = (width + 2 * padding - kernel_w) / stride + 1;

    auto output = torch::zeros({batch_size, out_channels, out_h, out_w}, x.options());

    dim3 threads(16, 16);
    dim3 blocks((out_w + threads.x - 1) / threads.x, (out_h + threads.y - 1) / threads.y, out_channels);

    conv2d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        out_channels,
        height,
        width,
        kernel_h,
        kernel_w,
        stride,
        padding
    );

    if (bias.has_value()) {
        output.add_(bias.value().view({1, -1, 1, 1}));
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA 2D Convolution");
}