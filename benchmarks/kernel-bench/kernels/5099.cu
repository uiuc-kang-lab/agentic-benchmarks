#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ATen/AccumulateType.h>

// Constant memory for weight and bias
__constant__ float const_weight[1024];
__constant__ float const_bias[1024];

// Optimized CUDA kernel for LayerNorm forward using constant memory for weight and bias.
template <typename scalar_t>
__global__ void layernorm_forward_kernel_constant(
    const scalar_t* __restrict__ input,
    const float eps,
    scalar_t* __restrict__ output,
    const int normalized_size) {

  const int instance_idx = blockIdx.x;
  const int tid = threadIdx.x;
  
  // Align pointers to 128-bit boundary
  const scalar_t* __restrict__ in_ptr = input + instance_idx * normalized_size;
  scalar_t* __restrict__ out_ptr = output + instance_idx * normalized_size;

  using accscalar_t = at::acc_type<scalar_t, true>;
  
  // Shared memory
  extern __shared__ char smem[];
  accscalar_t* s_sum = reinterpret_cast<accscalar_t*>(smem);
  accscalar_t* s_sum_sq = s_sum + blockDim.x;

  accscalar_t local_sum = 0;
  accscalar_t local_sum_sq = 0;

  // Calculate number of float4 elements we can process
  const int vector_size = 4;
  const int aligned_size = normalized_size / vector_size * vector_size;
  
  // Process elements in chunks of 4 (float4) for coalesced and vectorized access
  for (int i = tid * vector_size; i < aligned_size; i += blockDim.x * vector_size) {
    float4 in_vec = *reinterpret_cast<const float4*>(&in_ptr[i]);
    accscalar_t val1 = static_cast<accscalar_t>(in_vec.x);
    accscalar_t val2 = static_cast<accscalar_t>(in_vec.y);
    accscalar_t val3 = static_cast<accscalar_t>(in_vec.z);
    accscalar_t val4 = static_cast<accscalar_t>(in_vec.w);
    local_sum += val1 + val2 + val3 + val4;
    local_sum_sq += val1 * val1 + val2 * val2 + val3 * val3 + val4 * val4;
  }

  // Handle remaining elements
  for (int i = aligned_size + tid; i < normalized_size; i += blockDim.x) {
    accscalar_t val = static_cast<accscalar_t>(__ldg(&in_ptr[i]));
    local_sum += val;
    local_sum_sq += val * val;
  }

  s_sum[tid] = local_sum;
  s_sum_sq[tid] = local_sum_sq;
  __syncthreads();

  // Parallel reduction
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      s_sum[tid] += s_sum[tid + stride];
      s_sum_sq[tid] += s_sum_sq[tid + stride];
    }
    __syncthreads();
  }

  // Compute statistics
  __shared__ accscalar_t mean;
  __shared__ accscalar_t inv_std;
  if (tid == 0) {
    mean = s_sum[0] / static_cast<accscalar_t>(normalized_size);
    accscalar_t var = s_sum_sq[0] / static_cast<accscalar_t>(normalized_size) - mean * mean;
    inv_std = static_cast<accscalar_t>(1) / sqrt(var + static_cast<accscalar_t>(eps));
  }
  __syncthreads();

  // Process output in vectors of 4 when possible
  for (int i = tid * vector_size; i < aligned_size; i += blockDim.x * vector_size) {
    float4 in_vec = *reinterpret_cast<const float4*>(&in_ptr[i]);
    float4 out_vec;
    out_vec.x = static_cast<scalar_t>((static_cast<accscalar_t>(in_vec.x) - mean) * inv_std * 
                                     static_cast<accscalar_t>(const_weight[i]) + static_cast<accscalar_t>(const_bias[i]));
    out_vec.y = static_cast<scalar_t>((static_cast<accscalar_t>(in_vec.y) - mean) * inv_std * 
                                     static_cast<accscalar_t>(const_weight[i + 1]) + static_cast<accscalar_t>(const_bias[i + 1]));
    out_vec.z = static_cast<scalar_t>((static_cast<accscalar_t>(in_vec.z) - mean) * inv_std * 
                                     static_cast<accscalar_t>(const_weight[i + 2]) + static_cast<accscalar_t>(const_bias[i + 2]));
    out_vec.w = static_cast<scalar_t>((static_cast<accscalar_t>(in_vec.w) - mean) * inv_std * 
                                     static_cast<accscalar_t>(const_weight[i + 3]) + static_cast<accscalar_t>(const_bias[i + 3]));
    
    *reinterpret_cast<float4*>(&out_ptr[i]) = out_vec;
  }

  // Handle remaining elements
  for (int i = aligned_size + tid; i < normalized_size; i += blockDim.x) {
    scalar_t in_val = __ldg(&in_ptr[i]);
    accscalar_t norm_val = (static_cast<accscalar_t>(in_val) - mean) * inv_std;
    out_ptr[i] = static_cast<scalar_t>(norm_val * static_cast<accscalar_t>(const_weight[i]) + static_cast<accscalar_t>(const_bias[i]));
  }
}

// C++ interface function for the LayerNorm forward pass.
torch::Tensor layernorm_forward_constant(torch::Tensor x, torch::Tensor weight, torch::Tensor bias, double eps = 1e-5) {
  auto output = torch::empty_like(x);
  int normalized_size = weight.numel();
  int outer_size = x.numel() / normalized_size;

  // Copy weight and bias to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(const_weight), weight.data_ptr<float>(), normalized_size * sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(const_bias), bias.data_ptr<float>(), normalized_size * sizeof(float));

  // Ensure thread count is multiple of warp size (32)
  int threads = std::min(((normalized_size + 31) / 32) * 32, 1024);
  int blocks = outer_size;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "layernorm_forward_cuda", ([&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    int shared_size = threads * 2 * sizeof(accscalar_t);
    layernorm_forward_kernel_constant<scalar_t><<<blocks, threads, shared_size>>>(
        x.data_ptr<scalar_t>(),
        static_cast<float>(eps),
        output.data_ptr<scalar_t>(),
        normalized_size);
  }));

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &layernorm_forward_constant, "LayerNorm forward (CUDA) with constant memory",
        py::arg("x"), py::arg("weight"), py::arg("bias"), py::arg("eps") = 1e-5);
}
