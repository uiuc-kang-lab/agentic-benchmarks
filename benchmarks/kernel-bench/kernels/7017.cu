#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stdexcept>

template <typename scalar_t>
__global__ void argmin_kernel(const scalar_t* __restrict__ x,
                              int64_t* __restrict__ output,
                              int K,
                              int64_t outer_size,
                              int64_t inner_size) {
    extern __shared__ scalar_t shared_values[];
    
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t total_slices = outer_size * inner_size;
    if (idx >= total_slices) return;

    int64_t outer = idx / inner_size;
    int64_t inner = idx % inner_size;
    const scalar_t* slice_start = x + outer * (static_cast<int64_t>(K) * inner_size) + inner;

    scalar_t min_val = slice_start[0];
    int min_index = 0;

    const int tile_size = 32;
    for (int k_base = 1; k_base < K; k_base += tile_size) {
        int k_end = min(k_base + tile_size, K);
        
        // Load tile into shared memory
        for (int k = k_base; k < k_end; ++k) {
            shared_values[threadIdx.x * tile_size + (k - k_base)] = slice_start[k * inner_size];
        }
        __syncthreads();

        // Process tile from shared memory
        for (int k = k_base; k < k_end; ++k) {
            scalar_t val = shared_values[threadIdx.x * tile_size + (k - k_base)];
            if (val < min_val) {
                min_val = val;
                min_index = k;
            }
        }
        __syncthreads();
    }

    output[idx] = min_index;
}

at::Tensor argmin_cuda_forward(const at::Tensor &x, int64_t dim) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
    int dims = x.dim();
    if (dim < 0) dim += dims;
    TORCH_CHECK(dim >= 0 && dim < dims, "Reduction dim out of range");

    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) outer_size *= x.size(i);
    int K = static_cast<int>(x.size(dim));
    int64_t inner_size = 1;
    for (int i = dim + 1; i < dims; i++) inner_size *= x.size(i);

    std::vector<int64_t> out_sizes;
    for (int i = 0; i < dims; i++) if (i != dim) out_sizes.push_back(x.size(i));
    auto output = at::empty(out_sizes, x.options().dtype(at::kLong));

    int64_t total_slices = outer_size * inner_size;
    int threads = 256;
    int blocks = (total_slices + threads - 1) / threads;

    AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, x.scalar_type(), "argmin_cuda_forward", ([&] {
        size_t shared_mem = tile_size * threads * sizeof(scalar_t);
        argmin_kernel<scalar_t><<<blocks, threads, shared_mem>>>(
            x.data_ptr<scalar_t>(),
            output.data_ptr<int64_t>(),
            K,
            outer_size,
            inner_size
        );
    }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &argmin_cuda_forward, "Argmin forward (CUDA)");
}