#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <vector>

namespace py = pybind11;

__constant__ float c_weight[16384];

#define NUM_STREAMS 4
#define BLOCK_SIZE 256

__global__ void conv_transpose2d_forward_kernel(
    const float* __restrict__ input,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int N,
    const int C_in,
    const int H_in,
    const int W_in,
    const int C_out,
    const int H_out,
    const int W_out,
    const int kH,
    const int kW,
    const int sH,
    const int sW,
    const int pH,
    const int pW,
    const int batch_offset,
    const int batch_size
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size * C_out * H_out * W_out) return;

    const int ow = idx % W_out;
    const int oh = (idx / W_out) % H_out;
    const int oc = (idx / (W_out * H_out)) % C_out;
    const int n = batch_offset + (idx / (W_out * H_out * C_out));

    float sum = 0.0f;

    #pragma unroll
    for (int ic = 0; ic < C_in; ++ic) {
        #pragma unroll
        for (int kh = 0; kh < kH; ++kh) {
            #pragma unroll
            for (int kw = 0; kw < kW; ++kw) {
                const int i_val = oh + pH - kh;
                const int j_val = ow + pW - kw;

                if ((i_val % sH == 0) && (j_val % sW == 0)) {
                    const int i_in = i_val / sH;
                    const int j_in = j_val / sW;

                    if (i_in >= 0 && i_in < H_in && j_in >= 0 && j_in < W_in) {
                        const int input_idx = ((n * C_in + ic) * H_in + i_in) * W_in + j_in;
                        const int weight_idx = ((ic * C_out + oc) * kH + kh) * kW + kw;
                        sum += input[input_idx] * c_weight[weight_idx];
                    }
                }
            }
        }
    }

    if (bias != nullptr) {
        sum += bias[oc];
    }

    const int output_idx = ((n * C_out + oc) * H_out + oh) * W_out + ow;
    output[output_idx] = sum;
}

torch::Tensor conv_transpose2d_forward(
    torch::Tensor x,
    torch::Tensor weight,
    py::object bias_obj,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding
) {
    const int weight_size = weight.numel() * sizeof(float);
    if (weight_size > 64 * 1024) {
        c10::optional<torch::Tensor> bias = c10::nullopt;
        if (!bias_obj.is_none()) {
            bias = bias_obj.cast<torch::Tensor>();
        }
        return at::conv_transpose2d(x, weight, bias, stride, padding);
    }

    hipMemcpyToSymbol(HIP_SYMBOL(c_weight), weight.data_ptr<float>(), weight_size);

    torch::Tensor bias;
    const float* bias_ptr = nullptr;
    if (!bias_obj.is_none()) {
        bias = bias_obj.cast<torch::Tensor>();
        bias_ptr = bias.data_ptr<float>();
    }

    const int N = x.size(0);
    const int C_in = x.size(1);
    const int H_in = x.size(2);
    const int W_in = x.size(3);
    const int C_out = weight.size(1);
    const int kH = weight.size(2);
    const int kW = weight.size(3);
    const int sH = stride[0];
    const int sW = stride[1];
    const int pH = padding[0];
    const int pW = padding[1];

    const int H_out = (H_in - 1) * sH - 2 * pH + kH;
    const int W_out = (W_in - 1) * sW - 2 * pW + kW;

    auto output = torch::zeros({N, C_out, H_out, W_out}, x.options());

    // Create CUDA streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    // Calculate batch size per stream
    const int batch_per_stream = (N + NUM_STREAMS - 1) / NUM_STREAMS;
    const int elements_per_batch = C_out * H_out * W_out;

    // Process batches in parallel streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        const int batch_offset = i * batch_per_stream;
        const int current_batch_size = std::min(batch_per_stream, N - batch_offset);
        
        if (current_batch_size <= 0) continue;

        const int total_elements = current_batch_size * elements_per_batch;
        const int num_blocks = (total_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

        // Launch kernel in stream
        conv_transpose2d_forward_kernel<<<num_blocks, BLOCK_SIZE, 0, streams[i]>>>(
            x.data_ptr<float>(),
            bias_ptr,
            output.data_ptr<float>(),
            N, C_in, H_in, W_in,
            C_out, H_out, W_out,
            kH, kW, sH, sW, pH, pW,
            batch_offset,
            current_batch_size
        );
    }

    // Synchronize all streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward, "Conv Transpose 2D forward with stream pipelining",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride"),
          py::arg("padding"));
}