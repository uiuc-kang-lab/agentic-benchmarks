#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t, int BLOCK_SIZE=256, int TILE_SIZE=32>
__global__ void max_pool2d_kernel_optimized(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height, 
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    __shared__ scalar_t shared_input[TILE_SIZE][TILE_SIZE];
    
    const int tid = threadIdx.x;
    const int output_idx = blockIdx.x * blockDim.x + tid;
    
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    const int ow = output_idx % output_width;
    const int oh = (output_idx / output_width) % output_height;
    const int c = (output_idx / (output_width * output_height)) % channels;
    const int b = output_idx / (output_width * output_height * channels);

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    
    const int start_ih = oh * stride - padding;
    const int start_iw = ow * stride - padding;
    
    #pragma unroll
    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = start_ih + kh * dilation;
        
        #pragma unroll
        for (int kw = 0; kw < kernel_size; kw++) {
            const int iw = start_iw + kw * dilation;
            
            if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
                const int input_idx = b * (channels * input_height * input_width) +
                                    c * (input_height * input_width) +
                                    ih * input_width +
                                    iw;
                max_val = max(max_val, __ldg(&input[input_idx]));
            }
        }
    }

    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward_optimized(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool2d_cuda_forward_optimized", ([&] {
        max_pool2d_kernel_optimized<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}