#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32
#define VECTOR_SIZE 4

template<typename T>
__device__ __forceinline__ T load_broadcast_diagonal(const T* __restrict__ A, int warpId) {
    T a_val = __shfl_sync(0xffffffff, A[warpId], 0);
    return a_val;
}

template<typename T>
__device__ __forceinline__ void process_vectorized(
    const T* __restrict__ B,
    T* __restrict__ C,
    const T diag_val,
    const int row,
    const int vec_M,
    const int lane
) {
    using Vec4 = typename std::conditional<std::is_same<T, float>::value, float4, double4>::type;
    
    #pragma unroll
    for (int vec = lane; vec < vec_M; vec += WARP_SIZE) {
        int idx = row * vec_M + vec;
        Vec4 b_val = reinterpret_cast<const Vec4*>(B)[idx];
        Vec4 c_val;
        c_val.x = diag_val * b_val.x;
        c_val.y = diag_val * b_val.y;
        c_val.z = diag_val * b_val.z;
        c_val.w = diag_val * b_val.w;
        reinterpret_cast<Vec4*>(C)[idx] = c_val;
    }
}

template<typename T>
__device__ __forceinline__ void process_scalar(
    const T* __restrict__ B,
    T* __restrict__ C,
    const T diag_val,
    const int row_offset,
    const int M,
    const int lane
) {
    for (int col = lane; col < M; col += WARP_SIZE) {
        int idx = row_offset + col;
        C[idx] = diag_val * B[idx];
    }
}

__global__ void diag_matmul_kernel_modular(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    const int64_t N,
    const int64_t M
) {
    int lane = threadIdx.x % WARP_SIZE;
    int warpId = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;

    if (warpId < N) {
        const float diag_val = load_broadcast_diagonal(A, warpId);
        const int row = warpId;
        const int row_offset = row * M;

        if (M % VECTOR_SIZE == 0) {
            process_vectorized(B, C, diag_val, row, M/VECTOR_SIZE, lane);
        } else {
            process_scalar(B, C, diag_val, row_offset, M, lane);
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.dim() == 1, "A must be a 1D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == B.size(0), "Dimension mismatch: A.size(0) must match B.size(0)");

    A = A.contiguous();
    B = B.contiguous();

    int64_t N = A.size(0);
    int64_t M = B.size(1);

    auto C = torch::empty({N, M}, B.options());

    int threadsPerBlock = 128;
    int warpsPerBlock = threadsPerBlock / WARP_SIZE;
    int blocks = (N + warpsPerBlock - 1) / warpsPerBlock;

    diag_matmul_kernel_modular<<<blocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N,
        M
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Diagonal matrix multiplication using modular device functions");
}