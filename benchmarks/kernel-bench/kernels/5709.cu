#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void max_pool2d_coalesced_kernel(
    const scalar_t* input,
    scalar_t* output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int b = blockIdx.x;
    const int c = blockIdx.y;
    const int oh = blockIdx.z * blockDim.y + threadIdx.y;
    const int ow = threadIdx.x;

    if (b >= batch_size || c >= channels || oh >= output_height || ow >= output_width) return;

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    for (int kh = 0; kh < kernel_size; ++kh) {
        for (int kw = 0; kw < kernel_size; ++kw) {
            const int ih = oh * stride - padding + kh * dilation;
            const int iw = ow * stride - padding + kw * dilation;

            if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
                const int input_offset = b * channels * input_height * input_width
                                        + c * input_height * input_width
                                        + ih * input_width
                                        + iw;
                max_val = max(max_val, input[input_offset]);
            }
        }
    }

    const int output_offset = b * channels * output_height * output_width
                            + c * output_height * output_width
                            + oh * output_width
                            + ow;
    output[output_offset] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    constexpr int BLOCK_X = 32;
    constexpr int BLOCK_Y = 8;
    dim3 block(BLOCK_X, BLOCK_Y);
    dim3 grid(batch_size, channels, (output_height + BLOCK_Y - 1) / BLOCK_Y);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_coalesced_kernel<scalar_t><<<grid, block>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}
