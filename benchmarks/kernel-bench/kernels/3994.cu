#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void elu_kernel_vec4(const float4* x, float4* out, float alpha, int n4) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n4) {
        float4 val = x[idx];
        float4 result;
        
        result.x = (val.x > 0) ? val.x : alpha * (expf(val.x) - 1);
        result.y = (val.y > 0) ? val.y : alpha * (expf(val.y) - 1);
        result.z = (val.z > 0) ? val.z : alpha * (expf(val.z) - 1);
        result.w = (val.w > 0) ? val.w : alpha * (expf(val.w) - 1);
        
        out[idx] = result;
    }
}

torch::Tensor elu_cuda(torch::Tensor x, float alpha) {
    CHECK_INPUT(x);
    auto out = torch::empty_like(x);
    
    int n = x.numel();
    int n4 = n / 4;  // Number of float4 elements
    
    const int threads = 256;
    const int blocks = (n4 + threads - 1) / threads;
    
    // Handle the main part of the array with float4
    elu_kernel_vec4<<<blocks, threads>>>(
        reinterpret_cast<const float4*>(x.data_ptr<float>()),
        reinterpret_cast<float4*>(out.data_ptr<float>()),
        alpha,
        n4
    );
    
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &elu_cuda, "ELU activation (CUDA)");
}