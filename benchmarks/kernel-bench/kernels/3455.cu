#include "hip/hip_runtime.h"
/*
 * Optimized GELU activation with unified scalar and vectorized kernels
 * Supports both float and double types using appropriate vectorized loads/stores
 * and a fallback scalar kernel for remaining elements.
 */

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>

// Device function: GELU activation applied elementwise
// Specializations for float and double types

template <typename scalar_t>
__device__ inline scalar_t gelu_function(scalar_t x);

template <>
__device__ inline float gelu_function<float>(float x) {
    // GELU(x) = x * 0.5 * (1 + erf(x / sqrt(2)))
    return x * 0.5f * (1.0f + erff(x / 1.4142135623730951f));
}

template <>
__device__ inline double gelu_function<double>(double x) {
    return x * 0.5 * (1.0 + erf(x / 1.4142135623730951));
}

// Scalar kernel: processes elements one-by-one

template <typename scalar_t>
__global__ void gelu_kernel_scalar(const scalar_t* __restrict__ x,
                                     scalar_t* __restrict__ y,
                                     size_t numel) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numel) {
        scalar_t val = __ldg(&x[idx]);
        y[idx] = gelu_function<scalar_t>(val);
    }
}

// Vectorized kernel for float using float4: processes 4 floats at a time

__global__ void gelu_kernel_vectorized_float(const float* __restrict__ x,
                                               float* __restrict__ y,
                                               size_t num_vec) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_vec) {
        const float4* x_vec = reinterpret_cast<const float4*>(x);
        float4* y_vec = reinterpret_cast<float4*>(y);

        float4 in = x_vec[idx];
        float4 out;
        out.x = gelu_function<float>(in.x);
        out.y = gelu_function<float>(in.y);
        out.z = gelu_function<float>(in.z);
        out.w = gelu_function<float>(in.w);
        y_vec[idx] = out;
    }
}

// Vectorized kernel for double using double2: processes 2 doubles at a time

__global__ void gelu_kernel_vectorized_double(const double* __restrict__ x,
                                                double* __restrict__ y,
                                                size_t num_vec) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_vec) {
        const double2* x_vec = reinterpret_cast<const double2*>(x);
        double2* y_vec = reinterpret_cast<double2*>(y);

        double2 in = x_vec[idx];
        double2 out;
        out.x = gelu_function<double>(in.x);
        out.y = gelu_function<double>(in.y);
        y_vec[idx] = out;
    }
}

// Forward function callable from Python

torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
    
    auto output = torch::empty_like(x);
    size_t total = x.numel();
    const int threads = 1024;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "unified_gelu_cuda", ([&] {
        auto x_ptr = x.data_ptr<scalar_t>();
        auto y_ptr = output.data_ptr<scalar_t>();

        // Determine the vectorization factor: 4 for float, 2 for double
        int vec_factor = std::is_same<scalar_t, float>::value ? 4 : 2;

        // Check for proper alignment and whether we have enough elements to vectorize
        if (total >= (size_t)vec_factor &&
            (reinterpret_cast<uintptr_t>(x_ptr) % (sizeof(scalar_t) * vec_factor) == 0)) {
            size_t num_vec = total / vec_factor;
            size_t remainder = total % vec_factor;

            int blocks = (num_vec + threads - 1) / threads;
            if (num_vec > 0) {
                if (std::is_same<scalar_t, float>::value) {
                    gelu_kernel_vectorized_float<<<blocks, threads>>>(
                        reinterpret_cast<const float*>(x_ptr),
                        reinterpret_cast<float*>(y_ptr),
                        num_vec);
                } else { // double
                    gelu_kernel_vectorized_double<<<blocks, threads>>>(
                        reinterpret_cast<const double*>(x_ptr),
                        reinterpret_cast<double*>(y_ptr),
                        num_vec);
                }
            }
            // Process remaining elements with the scalar kernel
            if (remainder > 0) {
                int rem_blocks = (remainder + threads - 1) / threads;
                gelu_kernel_scalar<scalar_t><<<rem_blocks, threads>>>(
                    x_ptr + num_vec * vec_factor,
                    y_ptr + num_vec * vec_factor,
                    remainder);
            }
        } else {
            // Fallback: use scalar kernel if data is unaligned or not enough for vectorization
            int blocks = (total + threads - 1) / threads;
            gelu_kernel_scalar<scalar_t><<<blocks, threads>>>(x_ptr, y_ptr, total);
        }
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized vectorized GELU activation forward (CUDA)");
}
