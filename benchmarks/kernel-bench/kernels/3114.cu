#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#define THREADS_PER_BLOCK 256

// CUDA kernel declaration
__global__ void softmax_kernel(const float* __restrict__ x, float* __restrict__ y, int num_features);

// CUDA forward function
void softmax_forward_cuda(const float* x, float* y, int batch_size, int num_features) {
    dim3 block_dim(THREADS_PER_BLOCK);
    dim3 grid_dim((batch_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    int shared_mem_size = sizeof(float) * THREADS_PER_BLOCK;

    softmax_kernel<<<grid_dim, block_dim, shared_mem_size>>>(x, y, num_features);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in softmax_forward_cuda: %s\n", hipGetErrorString(err));
        return;
    }
}

__global__ void softmax_kernel(const float* __restrict__ x, float* __restrict__ y, int num_features) {
    int batch_idx = blockIdx.x;
    int tid = threadIdx.x;
    int stride = blockDim.x;

    const float* __restrict__ x_row = x + batch_idx * num_features;
    float* __restrict__ y_row = y + batch_idx * num_features;

    extern __shared__ float sdata[];

    float max_val = -INFINITY;
    for (int i = tid; i < num_features; i += stride) {
        float val = __ldg(&x_row[i]);
        max_val = max(max_val, val);
    }

    sdata[tid] = max_val;
    __syncthreads();

    for (unsigned int s = stride / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = max(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    max_val = sdata[0];

    float sum_val = 0.0f;
    for (int i = tid; i < num_features; i += stride) {
        float val = __ldg(&x_row[i]);
        float exp_val = __expf(val - max_val);
        y_row[i] = exp_val;
        sum_val += exp_val;
    }

    sdata[tid] = sum_val;
    __syncthreads();

    for (unsigned int s = stride / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    sum_val = sdata[0];

    for (int i = tid; i < num_features; i += stride) {
        y_row[i] = y_row[i] / sum_val;
    }
}

torch::Tensor forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor.");
    TORCH_CHECK(x.dim() == 2, "Input tensor must be 2D.");
    TORCH_CHECK(x.scalar_type() == torch::kFloat32, "Input tensor must be float32.");

    int batch_size = x.size(0);
    int num_features = x.size(1);

    auto y = torch::empty_like(x);

    softmax_forward_cuda(x.data_ptr<float>(), y.data_ptr<float>(), batch_size, num_features);

    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Softmax forward (CUDA)");
}
