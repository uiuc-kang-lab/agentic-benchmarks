#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/cuda/HIPContext.h>
#include <vector>

#define BLOCK_SIZE_STATS 512
#define BLOCK_SIZE_NORM 256
#define ELEMENTS_PER_THREAD 4

typedef float4 float4_t;

template <typename T>
__device__ __forceinline__ T warpReduceSum(T val) {
    #pragma unroll
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

template <typename T>
__device__ __forceinline__ T blockReduceSum(T val) {
    static __shared__ T shared[32];
    const int lid = threadIdx.x % warpSize;
    const int wid = threadIdx.x / warpSize;

    val = warpReduceSum(val);
    if (lid == 0) shared[wid] = val;
    __syncthreads();

    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lid] : 0;
    if (wid == 0) val = warpReduceSum(val);
    
    return val;
}

template <typename scalar_t>
__global__ void compute_stats_kernel(
    const scalar_t* __restrict__ x,
    const int N,
    const int C,
    const int spatial,
    const int channels_per_group,
    const int num_groups,
    scalar_t* __restrict__ mean,
    scalar_t* __restrict__ var) {

    const int idx = blockIdx.x;
    const int n = idx / num_groups;
    const int g = idx % num_groups;
    
    const int group_offset = n * C * spatial + g * channels_per_group * spatial;
    const int group_elems = channels_per_group * spatial;
    
    const int vec_size = sizeof(float4_t) / sizeof(scalar_t);
    const int num_vectors = group_elems / vec_size;
    const int remaining = group_elems % vec_size;
    
    scalar_t thread_sum = 0;
    scalar_t thread_sum_sq = 0;

    const float4_t* x_vec = reinterpret_cast<const float4_t*>(x + group_offset);
    #pragma unroll 4
    for (int i = threadIdx.x; i < num_vectors; i += blockDim.x) {
        float4_t v = __ldg(x_vec + i);
        thread_sum += v.x + v.y + v.z + v.w;
        thread_sum_sq += v.x * v.x + v.y * v.y + v.z * v.z + v.w * v.w;
    }

    if (threadIdx.x < remaining) {
        scalar_t val = __ldg(x + group_offset + num_vectors * vec_size + threadIdx.x);
        thread_sum += val;
        thread_sum_sq += val * val;
    }

    thread_sum = blockReduceSum(thread_sum);
    thread_sum_sq = blockReduceSum(thread_sum_sq);

    if (threadIdx.x == 0) {
        const scalar_t group_mean = thread_sum / group_elems;
        const scalar_t group_var = thread_sum_sq / group_elems - group_mean * group_mean;
        const int out_index = n * num_groups + g;
        mean[out_index] = group_mean;
        var[out_index] = group_var;
    }
}

template <typename scalar_t>
__global__ void group_norm_forward_kernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ mean,
    const scalar_t* __restrict__ var,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    const int N,
    const int C,
    const int spatial,
    const int channels_per_group,
    const int num_groups,
    const scalar_t eps,
    scalar_t* __restrict__ y) {

    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int total = N * C * spatial;

    #pragma unroll ELEMENTS_PER_THREAD
    for (int base_idx = tid * ELEMENTS_PER_THREAD; base_idx < total; base_idx += stride * ELEMENTS_PER_THREAD) {
        float4_t x_val = __ldg(reinterpret_cast<const float4_t*>(x + base_idx));
        float4_t result;

        #pragma unroll
        for (int i = 0; i < ELEMENTS_PER_THREAD; i++) {
            const int idx = base_idx + i;
            if (idx < total) {
                const int j = idx % spatial;
                const int temp = idx / spatial;
                const int c = temp % C;
                const int n = temp / C;
                const int g = c / channels_per_group;
                const int stats_index = n * num_groups + g;

                const scalar_t m = __ldg(mean + stats_index);
                const scalar_t v = __ldg(var + stats_index);
                const scalar_t inv_std = rsqrt(v + eps);
                const scalar_t w = __ldg(weight + c);
                const scalar_t b = __ldg(bias + c);

                (&result.x)[i] = ((&x_val.x)[i] - m) * inv_std * w + b;
            }
        }
        
        *reinterpret_cast<float4_t*>(y + base_idx) = result;
    }
}

torch::Tensor group_norm_forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    int64_t num_groups,
    double eps) {

    const int N = x.size(0);
    const int C = x.size(1);
    int spatial = 1;
    for (int i = 2; i < x.dim(); i++) {
        spatial *= x.size(i);
    }
    const int channels_per_group = C / num_groups;

    auto y = torch::empty_like(x);
    auto options = torch::TensorOptions().device(x.device()).dtype(x.dtype());
    auto mean = torch::empty({N, num_groups}, options);
    auto var = torch::empty({N, num_groups}, options);

    const int total_groups = N * num_groups;
    const dim3 blocks_stats(total_groups);

    const int blocks_norm = (N * C * spatial + BLOCK_SIZE_NORM * ELEMENTS_PER_THREAD - 1) / 
                          (BLOCK_SIZE_NORM * ELEMENTS_PER_THREAD);

    hipStream_t stream = c10::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "group_norm_forward_cuda", ([&] {
        compute_stats_kernel<scalar_t><<<blocks_stats, BLOCK_SIZE_STATS, 0, stream>>>(
            x.data_ptr<scalar_t>(),
            N, C, spatial,
            channels_per_group,
            num_groups,
            mean.data_ptr<scalar_t>(),
            var.data_ptr<scalar_t>());

        group_norm_forward_kernel<scalar_t><<<blocks_norm, BLOCK_SIZE_NORM, 0, stream>>>(
            x.data_ptr<scalar_t>(),
            mean.data_ptr<scalar_t>(),
            var.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            bias.data_ptr<scalar_t>(),
            N, C, spatial,
            channels_per_group,
            num_groups,
            static_cast<scalar_t>(eps),
            y.data_ptr<scalar_t>());
    }));

    return y;
}