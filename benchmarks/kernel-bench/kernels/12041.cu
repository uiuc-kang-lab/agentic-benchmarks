#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__inline__ __device__ float warp_reduce_sum(float val) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__global__ void hinge_loss_warp_optimized_kernel(const float* __restrict__ predictions, const float* __restrict__ targets, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (idx < n) {
        const float pred = __ldg(&predictions[idx]);
        const float target = __ldg(&targets[idx]);
        sum = fmaxf(0.0f, 1.0f - pred * target);
    }
    
    // Perform warp reduction
    sum = warp_reduce_sum(sum);

    // Write reduced sum to global memory
    if (threadIdx.x % warpSize == 0) {
        atomicAdd(output, sum);
    }
}

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    CHECK_INPUT(predictions);
    CHECK_INPUT(targets);

    int n = predictions.numel();
    torch::Tensor output = torch::zeros({1}, predictions.options());

    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    hinge_loss_warp_optimized_kernel<<<blocks, threads>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    // Compute the mean of the output tensor
    auto mean = output / n;
    return mean;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Hinge Loss Warp Optimized Forward");
}