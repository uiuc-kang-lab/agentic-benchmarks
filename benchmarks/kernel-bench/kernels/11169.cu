#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Kernel that uses __ldg() for read-only accesses and leverages 128-bit aligned loads via float4
// to optimize global memory access.
__global__ void smooth_l1_loss_vec_ldg_kernel(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    int n_elements
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    float thread_sum = 0.0f;

    // Process data in chunks of 4 elements using float4 for 128-bit aligned memory accesses
    int vec_count = n_elements / 4;
    const float4* pred4 = reinterpret_cast<const float4*>(predictions);
    const float4* targ4 = reinterpret_cast<const float4*>(targets);

    for (int i = idx; i < vec_count; i += stride) {
        float4 p = __ldg(pred4 + i);
        float4 t = __ldg(targ4 + i);

        float diff = p.x - t.x;
        float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;

        diff = p.y - t.y;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;

        diff = p.z - t.z;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;

        diff = p.w - t.w;
        abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;
    }

    // Process any remaining elements using scalar loads
    int scalar_start = vec_count * 4;
    for (int i = scalar_start + idx; i < n_elements; i += stride) {
        float diff = __ldg(predictions + i) - __ldg(targets + i);
        float abs_diff = fabsf(diff);
        thread_sum += (abs_diff < 1.0f) ? 0.5f * diff * diff : abs_diff - 0.5f;
    }

    // Block-level reduction using shared memory
    __shared__ float smem[256];
    int tid = threadIdx.x;
    smem[tid] = thread_sum;
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        // Divide the total loss by n_elements for averaging
        atomicAdd(output, smem[0] / n_elements);
    }
}

// Host function wrapper
torch::Tensor smooth_l1_loss_vec_ldg(
    torch::Tensor predictions,
    torch::Tensor targets
) {
    TORCH_CHECK(
        predictions.sizes() == targets.sizes(),
        "Input tensors must have the same shape"
    );
    TORCH_CHECK(
        predictions.is_contiguous() && targets.is_contiguous(),
        "Input tensors must be contiguous"
    );
    TORCH_CHECK(
        predictions.device().is_cuda() && targets.device().is_cuda(),
        "Inputs must be CUDA tensors"
    );

    int n = predictions.numel();
    auto output = torch::zeros({1}, predictions.options());

    const int block_size = 256;
    int grid_size = (n / 4 + block_size - 1) / block_size;
    grid_size = grid_size > 0 ? grid_size : 1;

    smooth_l1_loss_vec_ldg_kernel<<<grid_size, block_size>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &smooth_l1_loss_vec_ldg, "Vectorized Smooth L1 Loss using __ldg and 128-bit aligned memory accesses");
}
