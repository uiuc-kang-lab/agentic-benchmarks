#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <c10/util/Optional.h>
#include <stdio.h>

// Inline device function to compute greatest common divisor
__device__ int gcd(int a, int b) {
  while (b != 0) {
    int t = b;
    b = a % b;
    a = t;
  }
  return a;
}

// Inline device function for minimum
__device__ int my_min(int a, int b) {
  return a < b ? a : b;
}

// Optimized CUDA kernel for 2D transposed convolution with loop unrolling
// This kernel is identical to previous optimized version with loop unrolling
__global__ void conv_transpose2d_kernel_unrolled(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int batch,
    const int in_channels,
    const int in_h,
    const int in_w,
    const int out_channels,
    const int out_h,
    const int out_w,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_h,
    const int pad_w,
    const int dilation_h,
    const int dilation_w,
    const int groups,
    const int in_channels_per_group,
    const int out_channels_per_group) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch * out_channels * out_h * out_w;
  if (index >= total) return;

  // Decode flat index into (n, oc, oh, ow) where n is relative to the sub-batch
  int ow = index % out_w;
  int tmp = index / out_w;
  int oh = tmp % out_h;
  tmp = tmp / out_h;
  int oc = tmp % out_channels;
  int n = tmp / out_channels;

  float out_val = bias[oc];

  // Determine which group this output channel belongs to
  int g = oc / out_channels_per_group;

  // Precompute candidate positions (output coordinate plus padding)
  int candidate_h = oh + pad_h;
  int candidate_w = ow + pad_w;

  // For the height dimension, compute the valid starting offset and step
  int offset_kh = -1;
  int mod_h = candidate_h % stride_h;
  for (int k = 0; k < stride_h; k++) {
    if ((k * dilation_h) % stride_h == mod_h) {
      offset_kh = k;
      break;
    }
  }
  int step_kh = stride_h / gcd(stride_h, dilation_h);
  int kh_bound = candidate_h / dilation_h + 1;
  int kh_end = my_min(kernel_h, kh_bound);

  // For the width dimension, compute the valid starting offset and step
  int offset_kw = -1;
  int mod_w = candidate_w % stride_w;
  for (int k = 0; k < stride_w; k++) {
    if ((k * dilation_w) % stride_w == mod_w) {
      offset_kw = k;
      break;
    }
  }
  int step_kw = stride_w / gcd(stride_w, dilation_w);
  int kw_bound = candidate_w / dilation_w + 1;
  int kw_end = my_min(kernel_w, kw_bound);

  // Iterate over the valid kernel positions in the height dimension with unrolling
  #pragma unroll
  for (int kh = offset_kh; kh >= 0 && kh < kh_end; kh += step_kh) {
    int h_in_candidate = candidate_h - kh * dilation_h;
    int ih = h_in_candidate / stride_h;
    if (ih < 0 || ih >= in_h) continue;

    // Iterate over the valid kernel positions in the width dimension with unrolling
    #pragma unroll
    for (int kw = offset_kw; kw >= 0 && kw < kw_end; kw += step_kw) {
      int w_in_candidate = candidate_w - kw * dilation_w;
      int iw = w_in_candidate / stride_w;
      if (iw < 0 || iw >= in_w) continue;

      // Loop over the corresponding input channels within the group
      #pragma unroll
      for (int c = g * in_channels_per_group; c < (g + 1) * in_channels_per_group; c++) {
        int x_index = n * (in_channels * in_h * in_w) +
                      c * (in_h * in_w) +
                      ih * in_w + iw;

        int weight_index = c * (out_channels_per_group * kernel_h * kernel_w) +
                           (oc - g * out_channels_per_group) * (kernel_h * kernel_w) +
                           kh * kernel_w + kw;

        out_val += x[x_index] * weight[weight_index];
      }
    }
  }

  int out_index = n * (out_channels * out_h * out_w) +
                  oc * (out_h * out_w) +
                  oh * out_w + ow;
  output[out_index] = out_val;
}

// Host function that wraps the CUDA kernel using multiple streams to overlap computation and memory transfers
at::Tensor forward(
    at::Tensor x,
    at::Tensor weight,
    c10::optional<at::Tensor> bias,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> dilation,
    int groups) {

  // Ensure inputs are contiguous
  x = x.contiguous();
  weight = weight.contiguous();
  if (bias.has_value() && bias.value().defined())
    bias = bias.value().contiguous();

  // Get dimensions from input and weight
  const int batch = x.size(0);
  const int in_channels = x.size(1);
  const int in_h = x.size(2);
  const int in_w = x.size(3);

  const int kernel_h = weight.size(2);
  const int kernel_w = weight.size(3);
  const int out_channels_per_group = weight.size(1);
  const int out_channels = out_channels_per_group * groups;

  // Retrieve convolution parameters
  const int stride_h = stride[0];
  const int stride_w = stride[1];
  const int pad_h = padding[0];
  const int pad_w = padding[1];
  const int dilation_h = dilation[0];
  const int dilation_w = dilation[1];

  // Compute output dimensions for conv_transpose2d
  const int out_h = (in_h - 1) * stride_h - 2 * pad_h + dilation_h * (kernel_h - 1) + 1;
  const int out_w = (in_w - 1) * stride_w - 2 * pad_w + dilation_w * (kernel_w - 1) + 1;

  // If bias was not provided, create a zero tensor
  if (!bias.has_value() || !bias.value().defined()) {
    bias = at::zeros({out_channels}, weight.options());
  }

  auto output = at::zeros({batch, out_channels, out_h, out_w}, x.options());

  int in_channels_per_group = in_channels / groups;

  // Determine number of streams to use (e.g. 4 or less if batch is small)
  int num_streams = (batch < 4) ? batch : 4;
  int batch_per_stream = (batch + num_streams - 1) / num_streams;

  // Create CUDA streams
  std::vector<hipStream_t> streams(num_streams);
  for (int i = 0; i < num_streams; i++) {
    hipStreamCreate(&streams[i]);
  }

  // Launch kernels for each sub-batch asynchronously using different streams
  for (int i = 0; i < num_streams; i++) {
    int start = i * batch_per_stream;
    if (start >= batch) break;
    int current_batch = ((start + batch_per_stream) <= batch) ? batch_per_stream : (batch - start);

    int total_threads = current_batch * out_channels * out_h * out_w;
    const int threads = 256;
    int blocks = (total_threads + threads - 1) / threads;

    // Compute pointer offsets for the current sub-batch
    const float* x_ptr = x.data_ptr<float>() + start * in_channels * in_h * in_w;
    float* output_ptr = output.data_ptr<float>() + start * out_channels * out_h * out_w;

    conv_transpose2d_kernel_unrolled<<<blocks, threads, 0, streams[i]>>>(
        x_ptr,
        weight.data_ptr<float>(),
        bias.value().data_ptr<float>(),
        output_ptr,
        current_batch, // sub-batch size
        in_channels,
        in_h,
        in_w,
        out_channels,
        out_h,
        out_w,
        kernel_h,
        kernel_w,
        stride_h,
        stride_w,
        pad_h,
        pad_w,
        dilation_h,
        dilation_w,
        groups,
        in_channels_per_group,
        out_channels_per_group
    );
  }

  // Synchronize and destroy all streams
  for (int i = 0; i < num_streams; i++) {
    hipStreamSynchronize(streams[i]);
    hipStreamDestroy(streams[i]);
  }

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "2D Transposed Convolution with Stream Overlap (CUDA)",
        py::arg("x"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride"),
        py::arg("padding"),
        py::arg("dilation"),
        py::arg("groups"));
}
