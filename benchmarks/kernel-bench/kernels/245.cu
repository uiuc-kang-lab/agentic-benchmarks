#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32
#define WARP_SIZE 32

__global__ void bmm_coalesced_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int batch_size,
    int M,
    int K,
    int N
) {
    // Shared memory with padding to avoid bank conflicts
    __shared__ float As[TILE_SIZE][TILE_SIZE + 1];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE + 1];
    
    int b = blockIdx.z;
    // Reorganize thread mapping for coalesced memory access
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    // Calculate base pointers for current batch
    const float* batch_A = A + b * M * K;
    const float* batch_B = B + b * K * N;
    float* batch_C = C + b * M * N;
    
    float sum = 0.0f;
    
    // Process tiles with coalesced memory access
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Coalesced loading of A: threads in a warp read consecutive elements
        if (row < M && t * TILE_SIZE + threadIdx.x < K) {
            As[threadIdx.y][threadIdx.x] = batch_A[row * K + t * TILE_SIZE + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        // Coalesced loading of B: threads in a warp read consecutive elements
        if (t * TILE_SIZE + threadIdx.y < K && col < N) {
            Bs[threadIdx.y][threadIdx.x] = batch_B[(t * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute partial products
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        
        __syncthreads();
    }
    
    // Coalesced storing of results: threads in a warp write to consecutive locations
    if (row < M && col < N) {
        // Ensure coalesced writes by having consecutive threads write to consecutive memory locations
        batch_C[row * N + col] = sum;
    }
}

torch::Tensor forward_bmm(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 3, "A must be 3D");
    TORCH_CHECK(B.dim() == 3, "B must be 3D");
    TORCH_CHECK(A.size(0) == B.size(0), "Batch sizes must match");
    TORCH_CHECK(A.size(2) == B.size(1), "Inner dimensions (K) must match");

    int batch_size = A.size(0);
    int M = A.size(1);
    int K = A.size(2);
    int N = B.size(2);

    auto options = torch::TensorOptions().dtype(A.dtype()).device(A.device());
    auto C = torch::zeros({batch_size, M, N}, options);

    // Configure grid and block dimensions for coalesced access
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, 
              (M + TILE_SIZE - 1) / TILE_SIZE, 
              batch_size);

    bmm_coalesced_kernel<<<grid, block>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        batch_size, M, K, N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_bmm, "Batched matrix multiplication with coalesced memory access (CUDA)");
}