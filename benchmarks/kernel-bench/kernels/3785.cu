#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_softplus(const scalar_t x) {
    if (x > static_cast<scalar_t>(20.0)) {
        return x;
    } else if (x < static_cast<scalar_t>(-20.0)) {
        return exp(x);
    }
    return log1p(exp(x));
}

template <typename scalar_t>
__global__ void softplus_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    extern __shared__ char shared_memory[];
    scalar_t* shared_data = reinterpret_cast<scalar_t*>(shared_memory);
    
    const int tid = threadIdx.x;
    const int block_size = blockDim.x;
    int idx = blockIdx.x * block_size + tid;
    const int grid_stride = gridDim.x * block_size;
    
    while (idx < size) {
        // Load data into shared memory
        shared_data[tid] = (idx < size) ? input[idx] : 0;
        __syncthreads();  // Ensure shared memory is loaded
        
        // Process data from shared memory
        if (idx < size) {
            output[idx] = compute_softplus(shared_data[tid]);
        }
        
        idx += grid_stride;
    }
}

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    const int threads = 256;
    const int blocks = (size + threads - 1) / threads;
    const int shared_memory_size = threads * sizeof(float);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "softplus_forward_cuda", ([&] {
        softplus_kernel<scalar_t><<<blocks, threads, shared_memory_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}