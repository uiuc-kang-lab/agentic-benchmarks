#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void cross_entropy_loss_stride_kernel(
    const float* __restrict__ logits,
    const int64_t* __restrict__ targets,
    float* __restrict__ losses,
    int batch_size,
    int num_classes
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Stride loop for handling large workloads
    for (int i = tid; i < batch_size; i += stride) {
        const float* logits_i = logits + i * num_classes;
        int64_t target = targets[i];

        float max_logit = logits_i[0];
        for (int j = 1; j < num_classes; ++j) {
            max_logit = fmaxf(max_logit, logits_i[j]);
        }

        float sum_exp = 0.0f;
        for (int j = 0; j < num_classes; ++j) {
            sum_exp += expf(logits_i[j] - max_logit);
        }

        float log_sum_exp = logf(sum_exp);
        losses[i] = -(logits_i[target] - max_logit - log_sum_exp);
    }
}

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.is_cuda(), "predictions must be a CUDA tensor");
    TORCH_CHECK(targets.is_cuda(), "targets must be a CUDA tensor");
    TORCH_CHECK(predictions.dim() == 2, "predictions must be a 2D tensor");
    TORCH_CHECK(targets.dim() == 1, "targets must be a 1D tensor");

    int batch_size = predictions.size(0);
    int num_classes = predictions.size(1);
    auto losses = torch::empty({batch_size}, predictions.options());

    int threads = 256;
    int blocks = (batch_size + threads - 1) / threads;

    cross_entropy_loss_stride_kernel<<<blocks, threads>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<int64_t>(),
        losses.data_ptr<float>(),
        batch_size,
        num_classes
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA error: ", hipGetErrorString(err));

    return losses.mean();
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CrossEntropyLoss with stride loop optimization (CUDA)");
}