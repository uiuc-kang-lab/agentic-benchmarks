#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device function using compile-time type selection to minimize divergence
// and perform branchless clamping
template <typename scalar_t>
__device__ inline scalar_t clamp_val(scalar_t x) {
  if constexpr (std::is_same<scalar_t, float>::value) {
    return fminf(fmaxf(x, 0.f), 1.f);
  } else {
    return fmin(fmax(x, static_cast<scalar_t>(0)), static_cast<scalar_t>(1));
  }
}

// Combined kernel that applies HardSigmoid: y = clamp((x + 3) / 6, 0, 1)
// Utilizes branchless operations via the inline clamp and reduces warp divergence
// while overlapping memory operations with CUDA streams for efficiency

template <typename scalar_t>
__global__ void streamlined_hardsigmoid_kernel(const scalar_t* __restrict__ input,
                                               scalar_t* __restrict__ output,
                                               size_t numel) {
  // Calculate global thread index and stride
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = blockDim.x * gridDim.x;
  
  // Precompute constants as constexpr to avoid redundant computations
  constexpr scalar_t add_const = static_cast<scalar_t>(3);
  constexpr scalar_t div_const = static_cast<scalar_t>(1) / static_cast<scalar_t>(6);

  // Process elements in a stride loop for better load balancing among warps
  for (size_t i = idx; i < numel; i += stride) {
    scalar_t x = input[i];
    scalar_t y = (x + add_const) * div_const;
    // Branchless clamping using our inline helper
    y = clamp_val(y);
    output[i] = y;
  }
}

// Host function to launch the kernel with CUDA streams

torch::Tensor forward(torch::Tensor input) {
  TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");
  auto output = torch::empty_like(input);
  size_t numel = input.numel();
  const int threads = 1024;
  const int blocks = (numel + threads - 1) / threads;

  hipStream_t stream;
  hipStreamCreate(&stream);

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "streamlined_hardsigmoid_cuda", ([&] {
    streamlined_hardsigmoid_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
      input.data_ptr<scalar_t>(),
      output.data_ptr<scalar_t>(),
      numel);
  }));

  hipError_t err = hipGetLastError();
  TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "HardSigmoid activation forward (CUDA) with streams");
}