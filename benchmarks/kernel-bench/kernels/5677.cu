#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void max_pool2d_coalesced_block_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int ow = blockIdx.x * blockDim.x + threadIdx.x;
    const int oh = blockIdx.y * blockDim.y + threadIdx.y;
    const int bc = blockIdx.z;
    
    if (ow >= output_width || oh >= output_height || bc >= batch_size * channels) return;
    
    const int b = bc / channels;
    const int c = bc % channels;
    
    const int input_batch_offset = b * channels * input_height * input_width + c * input_height * input_width;
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = oh * stride - padding + kh * dilation;
        if (ih >= 0 && ih < input_height) {
            const int ih_offset = ih * input_width;
            
            #pragma unroll
            for (int kw = 0; kw < kernel_size; kw++) {
                const int iw = ow * stride - padding + kw * dilation;
                if (iw >= 0 && iw < input_width) {
                    max_val = fmaxf(max_val, __ldg(&input[input_batch_offset + ih_offset + iw]));
                }
            }
        }
    }

    const int output_idx = bc * output_height * output_width + oh * output_width + ow;
    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const dim3 block(32, 8);  // Aligned for 256 threads per block (32x8)
    const dim3 grid(
        (output_width + block.x - 1) / block.x,
        (output_height + block.y - 1) / block.y,
        batch_size * channels
    );

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_coalesced_block_kernel<scalar_t><<<grid, block>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward with 2D block coalescing (CUDA)");
}