#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void gelu_kernel(const float* x, float* y, int n) {
    const float sqrt_2_over_pi = 0.7978845608f;
    const float coeff = 0.044715f;
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for(int i = tid; i < n; i += stride) {
        float xi = x[i];
        float x_cubed = xi * xi * xi;
        float inner = xi + coeff * x_cubed;
        inner *= sqrt_2_over_pi;
        float tanh_val = tanhf(inner);
        y[i] = 0.5f * xi * (1.0f + tanh_val);
    }
}

torch::Tensor gelu_forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(x.is_contiguous(), "Input tensor must be contiguous");
    
    auto y = torch::empty_like(x);
    int n = x.numel();
    
    const int threads = 512;
    const int max_blocks = 65535;
    int blocks = min((n + threads - 1) / threads, max_blocks);
    
    gelu_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        n
    );
    
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &gelu_forward, "GELU forward CUDA implementation");
}