#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

// CUDA kernel for performing a depthwise 2D convolution with an asymmetric kernel
// (kernel height > 1, kernel width = 1)
__global__ void depthwise_conv2d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch,
    int channels,
    int in_h,
    int in_w,
    int out_h,
    int out_w,
    int kernel_h,
    int stride,
    int padding,
    int dilation)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch * channels * out_h * out_w;
    if (index < total) {
        // Decode the flattened index into 4D coordinates: (b, c, oh, ow)
        int ow = index % out_w;
        int tmp = index / out_w;
        int oh = tmp % out_h;
        tmp = tmp / out_h;
        int c = tmp % channels;
        int b = tmp / channels;

        float sum = 0.f;
        // Loop over kernel height dimension
        for (int kh = 0; kh < kernel_h; ++kh) {
            int ih = oh * stride - padding + kh * dilation;
            // Since kernel width is 1, the input column is computed as:
            int iw = ow * stride - padding;
            if (ih >= 0 && ih < in_h && iw >= 0 && iw < in_w) {
                int input_idx = ((b * channels + c) * in_h + ih) * in_w + iw;
                // weight shape: (channels, 1, kernel_h, 1) => index as (c, kh)
                int weight_idx = c * kernel_h + kh;
                sum += input[input_idx] * weight[weight_idx];
            }
        }
        // Add bias for the current channel
        sum += bias[c];
        int output_idx = ((b * channels + c) * out_h + oh) * out_w + ow;
        output[output_idx] = sum;
    }
}

// The forward function now accepts bias as an optional tensor.
// If bias is None, a zero bias tensor will be used.
at::Tensor forward(
    at::Tensor x,
    at::Tensor weight,
    c10::optional<at::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups)
{
    // Ensure the inputs are contiguous.
    x = x.contiguous();
    weight = weight.contiguous();

    // Retrieve input dimensions.
    int batch = x.size(0);
    int channels = x.size(1);
    int in_h = x.size(2);
    int in_w = x.size(3);
    int kernel_h = weight.size(2);  // weight shape: (channels, 1, kernel_h, 1)
    
    // For depthwise convolution, groups should equal the number of channels.
    if (groups != channels) {
        throw std::invalid_argument("Depthwise convolution requires groups == number of input channels.");
    }

    // Handle bias: if bias is None, create a zeros tensor.
    at::Tensor bias_val;
    if (bias.has_value() && bias.value().defined()) {
        bias_val = bias.value().contiguous();
    } else {
        bias_val = at::zeros({channels}, x.options());
    }

    // Compute output dimensions.
    int out_h = (in_h + 2 * padding - dilation * (kernel_h - 1) - 1) / stride + 1;
    int out_w = (in_w + 2 * padding - 1) / stride + 1;

    // Allocate output tensor.
    auto output = at::empty({batch, channels, out_h, out_w}, x.options());

    // Set up CUDA launch parameters.
    int total = batch * channels * out_h * out_w;
    int threads = 1024;
    int blocks = (total + threads - 1) / threads;

    const float* x_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias_val.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();

    // Launch the CUDA kernel.
    depthwise_conv2d_kernel<<<blocks, threads>>>(
        x_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch,
        channels,
        in_h,
        in_w,
        out_h,
        out_w,
        kernel_h,
        stride,
        padding,
        dilation
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Depthwise 2D Convolution forward (CUDA)",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = c10::nullopt,
          py::arg("stride"),
          py::arg("padding"),
          py::arg("dilation"),
          py::arg("groups"));
}