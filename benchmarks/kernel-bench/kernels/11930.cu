#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__device__ __forceinline__ scalar_t warp_reduce_sum(scalar_t val) {
    #pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

template <typename scalar_t>
__global__ void triplet_margin_loss_kernel(
    const scalar_t* __restrict__ anchor,
    const scalar_t* __restrict__ positive, 
    const scalar_t* __restrict__ negative,
    scalar_t* __restrict__ output,
    const float margin,
    const int batch_size,
    const int feat_size) {
    
    extern __shared__ scalar_t shared_mem[];
    
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int warp_id = tid / warpSize;
    const int lane_id = tid % warpSize;
    const int batch_idx = bid;
    
    if (batch_idx >= batch_size) return;
    
    scalar_t dist_pos = 0.0;
    scalar_t dist_neg = 0.0;
    
    #pragma unroll 4
    for (int feat_idx = tid; feat_idx < feat_size; feat_idx += blockDim.x) {
        const int idx = batch_idx * feat_size + feat_idx;
        const scalar_t a = anchor[idx];
        const scalar_t p = positive[idx];
        const scalar_t n = negative[idx];
        
        const scalar_t d_pos = a - p;
        const scalar_t d_neg = a - n;
        
        dist_pos += d_pos * d_pos;
        dist_neg += d_neg * d_neg;
    }
    
    dist_pos = warp_reduce_sum(dist_pos);
    dist_neg = warp_reduce_sum(dist_neg);
    
    if (lane_id == 0) {
        shared_mem[warp_id*2] = dist_pos;
        shared_mem[warp_id*2 + 1] = dist_neg;
    }
    
    __syncthreads();
    
    if (warp_id == 0 && lane_id < (blockDim.x/warpSize)) {
        dist_pos = shared_mem[lane_id*2];
        dist_neg = shared_mem[lane_id*2 + 1];
        
        dist_pos = warp_reduce_sum(dist_pos);
        dist_neg = warp_reduce_sum(dist_neg);
        
        if (lane_id == 0) {
            output[batch_idx] = max(scalar_t(0.0), sqrt(dist_pos) - sqrt(dist_neg) + margin);
        }
    }
}

torch::Tensor triplet_margin_loss_cuda(
    torch::Tensor anchor,
    torch::Tensor positive,
    torch::Tensor negative,
    float margin) {
    
    TORCH_CHECK(anchor.device().is_cuda(), "anchor must be a CUDA tensor");
    TORCH_CHECK(positive.device().is_cuda(), "positive must be a CUDA tensor");
    TORCH_CHECK(negative.device().is_cuda(), "negative must be a CUDA tensor");
    
    const int batch_size = anchor.size(0);
    const int feat_size = anchor.size(1);
    auto output = torch::zeros({batch_size}, anchor.options());
    
    const int block_size = 256;
    const int num_blocks = batch_size;
    const int shared_mem_size = (block_size/32) * 2 * sizeof(float);
    
    AT_DISPATCH_FLOATING_TYPES(anchor.scalar_type(), "triplet_margin_loss_kernel", ([&] {
        triplet_margin_loss_kernel<scalar_t><<<num_blocks, block_size, shared_mem_size>>>(
            anchor.data_ptr<scalar_t>(),
            positive.data_ptr<scalar_t>(),
            negative.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            margin,
            batch_size,
            feat_size);
    }));
    
    return output.mean();
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &triplet_margin_loss_cuda, "Triplet margin loss forward (CUDA)");
}