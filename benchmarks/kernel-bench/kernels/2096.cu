#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel to compute C = tril(A * B) for lower triangular matrices A and B,
// using __ldg() for read-only accesses and assuming 128-bit aligned memory (which torch typically ensures).
__global__ void triangular_mm_kernel_optimized(const float* __restrict__ A,
                                                const float* __restrict__ B,
                                                float* __restrict__ C,
                                                int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        // Only compute lower triangular part
        if (row < col) {
            C[row * N + col] = 0.0f;
        } else {
            float sum = 0.0f;
            // For lower triangular matrices, only k from col to row contribute.
            // Use __ldg() for read-only loads to take advantage of the read-only cache.
            for (int k = col; k <= row; ++k) {
                float a = __ldg(&A[row * N + k]);
                float b = __ldg(&B[k * N + col]);
                sum += a * b;
            }
            C[row * N + col] = sum;
        }
    }
}

// C++ interface exposed to PyTorch
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Define block and grid dimensions
    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threads - 1) / threads, (N + threads - 1) / threads);

    // Launch the optimized CUDA kernel
    triangular_mm_kernel_optimized<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized triangular matrix multiplication (CUDA) with __ldg and aligned memory accesses");
}
