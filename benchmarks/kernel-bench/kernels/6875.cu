#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <float.h>

__global__ void argmax_kernel_aligned(
    const float* __restrict__ x,
    int64_t* __restrict__ indices,
    const int outerSize,
    const int dimSize,
    const int innerSize) {

    int slice = blockIdx.x;
    if (slice >= outerSize * innerSize) return;

    int outer_idx = slice / innerSize;
    int inner_idx = slice % innerSize;
    int base_offset = outer_idx * (dimSize * innerSize) + inner_idx;

    float local_max = -FLT_MAX;
    int local_argmax = 0;

    // Use scalar loads with __ldg for correct computation
    for (int d = threadIdx.x; d < dimSize; d += blockDim.x) {
        float val = __ldg(&x[base_offset + d * innerSize]);
        if (val > local_max) {
            local_max = val;
            local_argmax = d;
        }
    }

    // Shared memory for reduction
    extern __shared__ char shared_mem[];
    float* __restrict__ s_max = reinterpret_cast<float*>(shared_mem);
    int* __restrict__ s_idx = reinterpret_cast<int*>(s_max + blockDim.x);

    s_max[threadIdx.x] = local_max;
    s_idx[threadIdx.x] = local_argmax;
    __syncthreads();

    // Optimized reduction using warp-level operations
    if (threadIdx.x < 64) {
        if (s_max[threadIdx.x + 64] > s_max[threadIdx.x]) {
            s_max[threadIdx.x] = s_max[threadIdx.x + 64];
            s_idx[threadIdx.x] = s_idx[threadIdx.x + 64];
        }
    }
    __syncthreads();

    if (threadIdx.x < 32) {
        // Warp-level reduction (no sync needed)
        volatile float* vmem = s_max;
        volatile int* vidx = s_idx;
        
        if (vmem[threadIdx.x + 32] > vmem[threadIdx.x]) {
            vmem[threadIdx.x] = vmem[threadIdx.x + 32];
            vidx[threadIdx.x] = vidx[threadIdx.x + 32];
        }
        if (vmem[threadIdx.x + 16] > vmem[threadIdx.x]) {
            vmem[threadIdx.x] = vmem[threadIdx.x + 16];
            vidx[threadIdx.x] = vidx[threadIdx.x + 16];
        }
        if (vmem[threadIdx.x + 8] > vmem[threadIdx.x]) {
            vmem[threadIdx.x] = vmem[threadIdx.x + 8];
            vidx[threadIdx.x] = vidx[threadIdx.x + 8];
        }
        if (vmem[threadIdx.x + 4] > vmem[threadIdx.x]) {
            vmem[threadIdx.x] = vmem[threadIdx.x + 4];
            vidx[threadIdx.x] = vidx[threadIdx.x + 4];
        }
        if (vmem[threadIdx.x + 2] > vmem[threadIdx.x]) {
            vmem[threadIdx.x] = vmem[threadIdx.x + 2];
            vidx[threadIdx.x] = vidx[threadIdx.x + 2];
        }
        if (vmem[threadIdx.x + 1] > vmem[threadIdx.x]) {
            vmem[threadIdx.x] = vmem[threadIdx.x + 1];
            vidx[threadIdx.x] = vidx[threadIdx.x + 1];
        }
    }

    if (threadIdx.x == 0) {
        indices[slice] = s_idx[0];
    }
}

torch::Tensor argmax_forward_cuda(const torch::Tensor& x, const int64_t dim) {
    TORCH_CHECK(x.scalar_type() == at::kFloat, "Only float32 is supported.");
    auto x_contig = x.contiguous();

    auto sizes = x_contig.sizes();
    int ndim = x_contig.dim();
    TORCH_CHECK(dim >= 0 && dim < ndim, "Invalid dim for argmax.");

    int outerSize = 1;
    for (int d = 0; d < dim; d++) {
        outerSize *= sizes[d];
    }
    int dimSize = sizes[dim];
    int innerSize = 1;
    for (int d = dim + 1; d < ndim; d++) {
        innerSize *= sizes[d];
    }

    std::vector<int64_t> out_sizes;
    for (int d = 0; d < ndim; d++) {
        if (d == dim) continue;
        out_sizes.push_back(sizes[d]);
    }
    auto options = torch::TensorOptions().device(x.device()).dtype(torch::kLong);
    auto indices = torch::empty(out_sizes, options);

    int slices = outerSize * innerSize;
    
    const int threads = 128;
    int blocks = slices;
    int sharedMemSize = threads * (sizeof(float) + sizeof(int));

    argmax_kernel_aligned<<<blocks, threads, sharedMemSize>>>(
        x_contig.data_ptr<float>(),
        indices.data_ptr<int64_t>(),
        outerSize,
        dimSize,
        innerSize
    );

    return indices;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &argmax_forward_cuda, "ArgMax CUDA forward with aligned memory access");
}