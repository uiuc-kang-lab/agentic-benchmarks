#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// This CUDA kernel uses warp-level primitives to reduce shared memory traffic for reductions
__global__ void cosine_similarity_loss_kernel_warp(const float* __restrict__ predictions,
                                                    const float* __restrict__ targets,
                                                    float* output,
                                                    int N,
                                                    int D) {
    // Each block processes one row
    int row = blockIdx.x;
    int tid = threadIdx.x;
    int blockSize = blockDim.x;

    const float* pred_row = predictions + row * D;
    const float* target_row = targets + row * D;

    float sum_dot = 0.0f;
    float sum_pred_sq = 0.0f;
    float sum_target_sq = 0.0f;

    // Loop over elements with stride equal to blockDim.x
    for (int i = tid; i < D; i += blockSize) {
        float p = pred_row[i];
        float t = target_row[i];
        sum_dot += p * t;
        sum_pred_sq += p * p;
        sum_target_sq += t * t;
    }

    // Warp-level reduction using __shfl_down_sync
    unsigned int mask = 0xffffffff;
    // Assume warp size of 32
    for (int offset = 16; offset > 0; offset /= 2) {
        sum_dot += __shfl_down_sync(mask, sum_dot, offset);
        sum_pred_sq += __shfl_down_sync(mask, sum_pred_sq, offset);
        sum_target_sq += __shfl_down_sync(mask, sum_target_sq, offset);
    }

    // Identify lane and warp ID
    int lane = tid & 31;        // tid % 32
    int warpId = tid >> 5;        // tid / 32
    int numWarps = (blockSize + warpSize - 1) / warpSize;

    // Allocate minimal shared memory for storing warp sums
    extern __shared__ float shared[];  // size: 3 * numWarps floats
    float* s_dot      = shared;
    float* s_pred_sq  = s_dot + numWarps;
    float* s_target_sq= s_pred_sq + numWarps;

    // Each warp's lane 0 writes its reduced sum to shared memory
    if (lane == 0) {
        s_dot[warpId] = sum_dot;
        s_pred_sq[warpId] = sum_pred_sq;
        s_target_sq[warpId] = sum_target_sq;
    }
    __syncthreads();

    // Final reduction: only threads in the first warp participate
    if (tid < numWarps) {
        sum_dot      = s_dot[tid];
        sum_pred_sq  = s_pred_sq[tid];
        sum_target_sq= s_target_sq[tid];
        
        // Use warp-level reduction if numWarps > 1
        for (int offset = (numWarps >> 1); offset > 0; offset /= 2) {
            sum_dot += __shfl_down_sync(0xffffffff, sum_dot, offset);
            sum_pred_sq += __shfl_down_sync(0xffffffff, sum_pred_sq, offset);
            sum_target_sq += __shfl_down_sync(0xffffffff, sum_target_sq, offset);
        }

        // Thread 0 in the first warp computes the final loss
        if (tid == 0) {
            const float eps = 1e-8f;
            float norm_pred = sqrtf(sum_pred_sq);
            float norm_target = sqrtf(sum_target_sq);
            float denominator = norm_pred * norm_target;
            denominator = fmaxf(denominator, eps);
            float cos_sim = sum_dot / denominator;
            atomicAdd(output, 1.0f - cos_sim);
        }
    }
}

// Host function that wraps the CUDA kernel
// This function performs necessary input checks and kernel launch

torch::Tensor cosine_similarity_loss_forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.dim() == 2, "predictions must be 2D");
    TORCH_CHECK(targets.dim() == 2, "targets must be 2D");
    TORCH_CHECK(predictions.sizes() == targets.sizes(), "Input tensors must have the same shape");
    TORCH_CHECK(predictions.scalar_type() == torch::kFloat32, "predictions must be float32");
    TORCH_CHECK(targets.scalar_type() == torch::kFloat32, "targets must be float32");

    int N = predictions.size(0);
    int D = predictions.size(1);

    auto output = torch::zeros({1}, predictions.options());
    const int block_size = 256;
    int numWarps = (block_size + 31) / 32;
    size_t shared_mem = 3 * numWarps * sizeof(float); // only needed for warp sums

    // Launch one block per sample
    cosine_similarity_loss_kernel_warp<<<N, block_size, shared_mem>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        N,
        D
    );

    output.div_(N);
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &cosine_similarity_loss_forward, "Cosine Similarity Loss Forward with warp-level primitives (CUDA)");
}
