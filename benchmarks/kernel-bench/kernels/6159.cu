#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__constant__ int const_params[3];  // kernel_size, stride, padding

template <typename scalar_t>
__global__ void avg_pool2d_forward_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int N,
    int C,
    int H,
    int W,
    int outH,
    int outW
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = N * C * outH * outW;
    if (index >= total) {
        return;
    }

    int w_out = index % outW;
    int h_out = (index / outW) % outH;
    int c = (index / (outW * outH)) % C;
    int n = index / (outW * outH * C);

    int h_start = h_out * const_params[1] - const_params[2];
    int w_start = w_out * const_params[1] - const_params[2];

    scalar_t sum_val = scalar_t(0);
    int h_in = max(0, h_start);
    int w_in = max(0, w_start);
    int h_end = min(H, h_start + const_params[0]);
    int w_end = min(W, w_start + const_params[0]);

    for (int i = h_in; i < h_end; i++) {
        for (int j = w_in; j < w_end; j++) {
            sum_val += input[((n * C + c) * H + i) * W + j];
        }
    }
    output[index] = sum_val / static_cast<scalar_t>(const_params[0] * const_params[0]);
}

torch::Tensor avg_pool2d_forward(
    torch::Tensor x,
    int kernel_size,
    int stride,
    int padding
) {
    TORCH_CHECK(x.dim() == 4, "Input must be a 4D tensor.");
    auto N = x.size(0);
    auto C = x.size(1);
    auto H = x.size(2);
    auto W = x.size(3);

    int outH = (H + 2 * padding - kernel_size) / stride + 1;
    int outW = (W + 2 * padding - kernel_size) / stride + 1;

    auto x_cont = x.contiguous();
    auto options = x.options();
    auto out = torch::empty({N, C, outH, outW}, options);

    const int threads = 128;
    const int blocks = (N * C * outH * outW + threads - 1) / threads;

    int h_params[3] = {kernel_size, stride, padding};
    hipMemcpyToSymbol(HIP_SYMBOL(const_params), h_params, sizeof(int) * 3);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "avg_pool2d_forward_kernel", ([&] {
        const scalar_t* input_data = x_cont.data_ptr<scalar_t>();
        scalar_t* output_data = out.data_ptr<scalar_t>();

        avg_pool2d_forward_kernel<<<blocks, threads>>>(
            input_data,
            output_data,
            N, C, H, W,
            outH, outW
        );
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err));

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &avg_pool2d_forward, "2D Average Pooling forward (CUDA)");
}
