#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel for batched matrix multiplication using warp-level primitives
// Shapes: A (batch_size, M, K), B (batch_size, K, N), C (batch_size, M, N)
__global__ void warp_bmm_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int batch_size,
    int M,
    int K,
    int N
) {
    int batch = blockIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float val = 0.0f;
        for (int k = 0; k < K; k++) {
            float a = A[batch * M * K + row * K + k];
            float b = B[batch * K * N + k * N + col];
            val += a * b;
        }

        // Use warp-level reduction to sum up the values
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            val += __shfl_down_sync(0xFFFFFFFF, val, offset);
        }

        // Write the result for the first thread in the warp
        if (threadIdx.x % warpSize == 0) {
            C[batch * M * N + row * N + col] = val;
        }
    }
}

torch::Tensor forward_warp_bmm(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 3, "A must be 3D");
    TORCH_CHECK(B.dim() == 3, "B must be 3D");
    TORCH_CHECK(A.size(0) == B.size(0), "Batch sizes must match");
    TORCH_CHECK(A.size(2) == B.size(1), "Inner dimensions (K) must match");

    int batch_size = A.size(0);
    int M = A.size(1);
    int K = A.size(2);
    int N = B.size(2);

    auto options = torch::TensorOptions().dtype(A.dtype()).device(A.device());
    auto C = torch::zeros({batch_size, M, N}, options);

    dim3 threads(32, 32);
    dim3 blocks((N + threads.x - 1) / threads.x, (M + threads.y - 1) / threads.y, batch_size);

    warp_bmm_kernel<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        batch_size, M, K, N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_warp_bmm, "Batched matrix multiplication with warp-level primitives (CUDA)");
}