#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16
#define WARP_SIZE 32

// CUDA kernel for batched matrix multiplication using warp-level tiling
// Computes C = A * B for each batch.
// A: (batch_size, M, K), B: (batch_size, K, N), C: (batch_size, M, N)
__global__ void bmm_warp_tiling_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int batch_size,
    int M,
    int K,
    int N
) {
    int b = blockIdx.z;  // Batch index
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float value = 0.0f;

    // Shared memory tiles for A and B
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int numTiles = (K + TILE_SIZE - 1) / TILE_SIZE;
    for (int t = 0; t < numTiles; t++) {
        // Load current tiles into shared memory
        if (row < M && (t * TILE_SIZE + threadIdx.x) < K) {
            As[threadIdx.y][threadIdx.x] = A[b * M * K + row * K + t * TILE_SIZE + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if ((t * TILE_SIZE + threadIdx.y) < K && col < N) {
            Bs[threadIdx.y][threadIdx.x] = B[b * K * N + (t * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        // Compute partial product for the tile
        for (int i = 0; i < TILE_SIZE; i++) {
            value += As[threadIdx.y][i] * Bs[i][threadIdx.x];
        }
        __syncthreads();
    }

    // Write the final computed value to C if within bounds
    if (row < M && col < N) {
        C[b * M * N + row * N + col] = value;
    }
}

// Forward function to launch the kernel
torch::Tensor forward_bmm(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 3, "A must be 3D");
    TORCH_CHECK(B.dim() == 3, "B must be 3D");
    TORCH_CHECK(A.size(0) == B.size(0), "Batch sizes must match");
    TORCH_CHECK(A.size(2) == B.size(1), "Inner dimensions (K) must match");

    int batch_size = A.size(0);
    int M = A.size(1);
    int K = A.size(2);
    int N = B.size(2);

    auto options = torch::TensorOptions().dtype(A.dtype()).device(A.device());
    torch::Tensor C = torch::zeros({batch_size, M, N}, options);

    // Configure grid and block dimensions
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE, batch_size);

    bmm_warp_tiling_kernel<<<grid, block>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        batch_size, M, K, N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_bmm, "Batched matrix multiplication with warp-level tiling (CUDA)");
}