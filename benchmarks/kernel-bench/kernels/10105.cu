#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <stdexcept>
#include <cstdio>

namespace py = pybind11;

#define THREADS_PER_BLOCK 256
// Define maximum number of elements in constant memory for depthwise weights
// Adjust this value if needed, but it must be within the hardware constant memory limits.
#define MAX_DEPTHWISE_WEIGHT_SIZE 16384

// Declare constant memory for depthwise weights for float and double types
__constant__ float depthwise_weight_const_f[MAX_DEPTHWISE_WEIGHT_SIZE];
__constant__ double depthwise_weight_const_d[MAX_DEPTHWISE_WEIGHT_SIZE];

// Template helper to get pointer to constant memory weights based on type
template<typename scalar_t>
__device__ __forceinline__ const scalar_t* get_depthwise_weight_const();

template<>
__device__ __forceinline__ const float* get_depthwise_weight_const<float>() {
    return depthwise_weight_const_f;
}

template<>
__device__ __forceinline__ const double* get_depthwise_weight_const<double>() {
    return depthwise_weight_const_d;
}

// Optimized depthwise convolution kernel using constant memory for weights.
// The weight data is read from constant memory via get_depthwise_weight_const<scalar_t>().
template <typename scalar_t>
__global__ void optimized_depthwise_conv2d_kernel(
    const scalar_t* __restrict__ input,   // [batch, channels, in_h, in_w]
    // Note: weight is now in constant memory, so it is not passed as a parameter.
    const scalar_t* __restrict__ bias,    // [channels] or nullptr
    scalar_t* __restrict__ output,        // [batch, channels, out_h, out_w]
    int batch,
    int channels,
    int in_h, int in_w,
    int out_h, int out_w,
    int k,
    int stride,
    int padding,
    int dilation) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch * channels * out_h * out_w;
  if (index >= total)
      return;

  // Decode flat index into (n, c, oh, ow)
  int ow = index % out_w;
  int tmp = index / out_w;
  int oh = tmp % out_h;
  tmp = tmp / out_h;
  int c = tmp % channels;
  int n = tmp / channels;

  scalar_t sum = 0;
  const scalar_t* weight_const = get_depthwise_weight_const<scalar_t>();

  // Loop over kernel dimensions and accumulate convolution result
  for (int i = 0; i < k; ++i) {
    for (int j = 0; j < k; ++j) {
      int ih = oh * stride - padding + i * dilation;
      int iw = ow * stride - padding + j * dilation;
      if (ih >= 0 && ih < in_h && iw >= 0 && iw < in_w) {
        int input_idx = n * channels * in_h * in_w + c * in_h * in_w + ih * in_w + iw;
        int weight_idx = c * k * k + i * k + j;
        sum += input[input_idx] * weight_const[weight_idx];
      }
    }
  }
  if (bias != nullptr)
    sum += bias[c];
  output[index] = sum;
}

// Pointwise (1x1) convolution kernel remains unchanged.
template <typename scalar_t>
__global__ void pointwise_conv2d_kernel(
    const scalar_t* __restrict__ input,   // [batch, in_channels, h, w]
    const scalar_t* __restrict__ weight,  // [out_channels, in_channels]
    const scalar_t* __restrict__ bias,    // [out_channels] or nullptr
    scalar_t* __restrict__ output,        // [batch, out_channels, h, w]
    int batch,
    int in_channels,
    int out_channels,
    int h,
    int w) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch * out_channels * h * w;
  if (index >= total)
      return;

  // Decode flat index into (n, oc, oh, ow)
  int ow = index % w;
  int tmp = index / w;
  int oh = tmp % h;
  tmp = tmp / h;
  int oc = tmp % out_channels;
  int n = tmp / out_channels;

  scalar_t sum = 0;
  for (int ic = 0; ic < in_channels; ++ic) {
    int input_idx = n * in_channels * h * w + ic * h * w + oh * w + ow;
    int weight_idx = oc * in_channels + ic;
    sum += input[input_idx] * weight[weight_idx];
  }
  if (bias != nullptr)
    sum += bias[oc];
  output[index] = sum;
}

// Core CUDA forward function with constant memory optimization for depthwise weights.
// It copies the depthwise weights to constant memory before launching the kernel.
torch::Tensor forward_cuda(
    const torch::Tensor& x,
    const torch::Tensor& depthwise_weight,
    const torch::Tensor& pointwise_weight,
    const torch::Tensor& depthwise_bias,
    const torch::Tensor& pointwise_bias,
    int stride,
    int padding,
    int dilation) {

  TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");
  TORCH_CHECK(depthwise_weight.is_cuda(), "Depthwise weight must be a CUDA tensor");
  TORCH_CHECK(pointwise_weight.is_cuda(), "Pointwise weight must be a CUDA tensor");
  if (depthwise_bias.defined() && depthwise_bias.numel() > 0)
      TORCH_CHECK(depthwise_bias.is_cuda(), "Depthwise bias must be a CUDA tensor if provided");
  if (pointwise_bias.defined() && pointwise_bias.numel() > 0)
      TORCH_CHECK(pointwise_bias.is_cuda(), "Pointwise bias must be a CUDA tensor if provided");

  int batch = x.size(0);
  int in_channels = x.size(1);
  int in_h = x.size(2);
  int in_w = x.size(3);

  // Depthwise weight is expected to have shape [in_channels, 1, k, k]
  int k = depthwise_weight.size(2);
  int out_h = (in_h + 2 * padding - dilation * (k - 1) - 1) / stride + 1;
  int out_w = (in_w + 2 * padding - dilation * (k - 1) - 1) / stride + 1;

  auto depthwise_output = torch::empty({batch, in_channels, out_h, out_w}, x.options());

  int total_depthwise = batch * in_channels * out_h * out_w;
  int threads = THREADS_PER_BLOCK;
  int blocks = (total_depthwise + threads - 1) / threads;

  // Before launching the depthwise kernel, copy the weights to constant memory.
  int weight_elements = in_channels * k * k;
  TORCH_CHECK(weight_elements <= MAX_DEPTHWISE_WEIGHT_SIZE, "Depthwise weight size exceeds constant memory limit");
  
  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "optimized_depthwise_conv2d_cuda", ([&] {
    size_t weight_size = weight_elements * sizeof(scalar_t);
    if (sizeof(scalar_t) == sizeof(float)) {
      hipMemcpyToSymbol(HIP_SYMBOL(depthwise_weight_const_f), depthwise_weight.data_ptr<scalar_t>(), weight_size, 0, hipMemcpyDeviceToDevice);
    } else {
      hipMemcpyToSymbol(HIP_SYMBOL(depthwise_weight_const_d), depthwise_weight.data_ptr<scalar_t>(), weight_size, 0, hipMemcpyDeviceToDevice);
    }

    optimized_depthwise_conv2d_kernel<scalar_t><<<blocks, threads>>>(
        x.data_ptr<scalar_t>(),
        reinterpret_cast<const scalar_t*>(
               (depthwise_bias.defined() && depthwise_bias.numel() > 0) ? depthwise_bias.data_ptr<scalar_t>() : nullptr),
        depthwise_output.data_ptr<scalar_t>(),
        batch,
        in_channels,
        in_h, in_w,
        out_h, out_w,
        k,
        stride,
        padding,
        dilation);
  }));
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Optimized depthwise kernel launch error: %s\n", hipGetErrorString(err));
  }

  // Pointwise convolution: weight shape is [out_channels, in_channels, 1, 1]
  int out_channels = pointwise_weight.size(0);
  auto output = torch::empty({batch, out_channels, out_h, out_w}, x.options());
  int total_pointwise = batch * out_channels * out_h * out_w;
  blocks = (total_pointwise + threads - 1) / threads;

  const void* pointwise_bias_ptr = (pointwise_bias.defined() && pointwise_bias.numel() > 0)
                                     ? pointwise_bias.data_ptr()
                                     : nullptr;

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "pointwise_conv2d_cuda", ([&] {
    pointwise_conv2d_kernel<scalar_t><<<blocks, threads>>>(
        depthwise_output.data_ptr<scalar_t>(),
        reinterpret_cast<const scalar_t*>(pointwise_weight.data_ptr<scalar_t>()),
        reinterpret_cast<const scalar_t*>(pointwise_bias_ptr),
        output.data_ptr<scalar_t>(),
        batch,
        in_channels,
        out_channels,
        out_h, out_w);
  }));
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Pointwise kernel launch error: %s\n", hipGetErrorString(err));
  }

  return output;
}

// Helper function: convert py::object to at::Tensor. If the object is None, returns an undefined tensor.
// If the object has a 'data' attribute (e.g., a torch.nn.Parameter), then that attribute is used.
at::Tensor toTensor(const py::object& obj) {
  if (obj.is_none()) {
    return at::Tensor();
  }
  try {
    return obj.cast<at::Tensor>();
  } catch (const py::cast_error& e) {
    if (py::hasattr(obj, "data")) {
      return obj.attr("data").cast<at::Tensor>();
    }
    throw std::runtime_error("Expected a torch Tensor or Parameter.");
  }
}

// Wrapper function to handle inputs that may be wrapped in Parameter objects or be None.
// Expected signature: forward(tensor, tensor, tensor, tensor, tensor, int, int, int) -> tensor
at::Tensor forward_wrapper(py::object x_obj,
                           py::object depthwise_weight_obj,
                           py::object pointwise_weight_obj,
                           py::object depthwise_bias_obj,
                           py::object pointwise_bias_obj,
                           int stride,
                           int padding,
                           int dilation) {

  auto x = toTensor(x_obj);
  auto depthwise_weight = toTensor(depthwise_weight_obj);
  auto pointwise_weight = toTensor(pointwise_weight_obj);
  auto depthwise_bias = toTensor(depthwise_bias_obj);
  auto pointwise_bias = toTensor(pointwise_bias_obj);

  return forward_cuda(x, depthwise_weight, pointwise_weight,
                      depthwise_bias, pointwise_bias,
                      stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward_wrapper, "Optimized CUDA depthwise separable convolution with constant memory");
}
