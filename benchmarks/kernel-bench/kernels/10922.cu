#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static const int BLOCK_SIZE = 256;

// Kernel function to compute MSE Loss
template <typename scalar_t>
__global__ void mse_forward_kernel(
    const scalar_t* __restrict__ preds,
    const scalar_t* __restrict__ tgts,
    double* __restrict__ sum_out,
    const int64_t num_elements
) {
    __shared__ double shm[BLOCK_SIZE];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double thread_sum = 0.0;

    // Strided loop to accumulate squared differences
    while (idx < num_elements) {
        double diff = static_cast<double>(preds[idx]) - static_cast<double>(tgts[idx]);
        thread_sum += diff * diff;
        idx += blockDim.x * gridDim.x;
    }

    // Store partial sums in shared memory
    shm[threadIdx.x] = thread_sum;
    __syncthreads();

    // Block-level reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shm[threadIdx.x] += shm[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Add the reduced sum from this block into global accumulator
    if (threadIdx.x == 0) {
        atomicAdd(sum_out, shm[0]);
    }
}

// Function to perform forward pass
torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.is_cuda(), "predictions must be a CUDA tensor");
    TORCH_CHECK(targets.is_cuda(), "targets must be a CUDA tensor");
    TORCH_CHECK(predictions.numel() == targets.numel(),
                "predictions and targets must have the same number of elements");

    const int64_t num_elements = predictions.numel();
    // Use double for accumulation
    auto accumulator = torch::zeros({1}, predictions.options().dtype(at::kDouble));

    const int grid_size = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Create CUDA stream for overlapping operations
    hipStream_t stream;
    hipStreamCreate(&stream);

    AT_DISPATCH_FLOATING_TYPES(predictions.scalar_type(), "mse_forward_cuda", [&] {
        mse_forward_kernel<scalar_t><<<grid_size, BLOCK_SIZE, 0, stream>>>(
            predictions.data_ptr<scalar_t>(),
            targets.data_ptr<scalar_t>(),
            accumulator.data_ptr<double>(),
            num_elements
        );
    });

    // Synchronize to ensure kernel execution completion
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    // Final mean = accumulator / N
    auto result = accumulator.div_(static_cast<double>(num_elements));
    return result.to(predictions.dtype());
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Mean Squared Error (MSE) forward (CUDA)");
}
