#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template<typename scalar_t, int VEC_SIZE>
__global__ void hardsigmoid_kernel(const scalar_t* __restrict__ input,
                                   scalar_t* __restrict__ output,
                                   size_t numel) {
    constexpr scalar_t three = 3.0;
    constexpr scalar_t sixth = 1.0/6.0;
    
    using vec_t = typename std::conditional<
        std::is_same<scalar_t, float>::value, float4,
        typename std::conditional<std::is_same<scalar_t, double>::value, double2, void>::type
    >::type;

    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int vec_tid = tid * VEC_SIZE;
    const int vec_stride = blockDim.x * gridDim.x * VEC_SIZE;

    for (int i = vec_tid; i < numel; i += vec_stride) {
        vec_t vec_in;
        scalar_t elements[VEC_SIZE];
        
        // Vectorized load
        *reinterpret_cast<vec_t*>(elements) = *reinterpret_cast<const vec_t*>(&input[i]);

        #pragma unroll
        for (int v = 0; v < VEC_SIZE; v++) {
            scalar_t x = elements[v];
            x = fma(x, sixth, three * sixth);  // (x + 3) / 6
            x = fmaxf(0.0f, fminf(1.0f, x));   // Built-in fast math functions
            elements[v] = x;
        }

        // Vectorized store
        *reinterpret_cast<vec_t*>(&output[i]) = *reinterpret_cast<vec_t*>(elements);
    }
}

torch::Tensor forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");
    auto output = torch::empty_like(input);
    const size_t numel = input.numel();
    
    constexpr int VEC_SIZE = sizeof(float4) / sizeof(float);  // 4 for float, 2 for double
    const int threads = 256;
    const int blocks = (numel + threads * VEC_SIZE - 1) / (threads * VEC_SIZE);

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "hardsigmoid_cuda", ([&] {
        hardsigmoid_kernel<scalar_t, VEC_SIZE><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            numel
        );
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "HardSigmoid activation forward (CUDA)");
}