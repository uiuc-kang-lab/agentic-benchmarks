#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/util/Optional.h>

namespace py = pybind11;

// ---------------------------------------------------------------------
// Optimized 1D convolution CUDA kernel using __ldg() for read-only accesses
// and assuming 128-bit aligned data for faster global memory loads.
// ---------------------------------------------------------------------
__global__ void conv1d_forward_kernel_opt(
    const float* __restrict__ x,
    const float* __restrict__ w,
    const float* __restrict__ bias_ptr, // may be null
    float* __restrict__ y,
    const int N,         // batch size
    const int C_in,      // input channels
    const int L_in,      // input length
    const int C_out,     // output channels
    const int K,         // kernel size
    const int stride,
    const int padding,
    const int dilation,
    const int groups,
    const int L_out      // output length
)
{
    // Each thread computes one output element: (n, out_ch, out_pos).
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = N * C_out * L_out;
    if (idx >= total) return;

    int out_pos = idx % L_out;
    int out_ch  = (idx / L_out) % C_out;
    int n       = idx / (L_out * C_out);

    // Determine group index based on channel grouping
    int group_size_out = C_out / groups;
    int group_size_in  = C_in  / groups;
    int group_idx      = out_ch / group_size_out;

    float val = 0.0f;
    // Convolution accumulation using __ldg() for read-only accesses
    for (int local_in_ch = 0; local_in_ch < group_size_in; local_in_ch++) {
        int in_ch = group_idx * group_size_in + local_in_ch;
        for (int k = 0; k < K; k++) {
            int in_pos = out_pos * stride + k * dilation - padding;
            if (in_pos >= 0 && in_pos < L_in) {
                int x_index = n * (C_in * L_in) + in_ch * L_in + in_pos;
                int w_index = out_ch * (group_size_in * K) + local_in_ch * K + k;
                float x_val = __ldg(&x[x_index]);
                float w_val = __ldg(&w[w_index]);
                val += x_val * w_val;
            }
        }
    }

    // Add bias if provided using __ldg()
    if (bias_ptr) {
        val += __ldg(&bias_ptr[out_ch]);
    }

    // Write result. Assumed to be 128-bit aligned when possible.
    int y_index = n * (C_out * L_out) + out_ch * L_out + out_pos;
    y[y_index] = val;
}

// ---------------------------------------------------------------------
// Implementation of conv1d forward with optional bias using the optimized kernel
// ---------------------------------------------------------------------
at::Tensor conv1d_forward_impl(
    const at::Tensor& x,
    const at::Tensor& weight,
    c10::optional<at::Tensor> bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups
)
{
    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.scalar_type() == at::kFloat, "x must be float32");
    TORCH_CHECK(weight.scalar_type() == at::kFloat, "weight must be float32");

    // x shape: [N, C_in, L_in]
    auto x_sizes = x.sizes();
    int64_t N    = x_sizes[0];
    int64_t C_in = x_sizes[1];
    int64_t L_in = x_sizes[2];

    // weight shape: [C_out, C_in/groups, K]
    auto w_sizes = weight.sizes();
    int64_t C_out = w_sizes[0];
    int64_t K     = w_sizes[2];

    // Calculate output length
    int64_t L_out = (L_in + 2 * padding - dilation * (K - 1) - 1) / stride + 1;
    TORCH_CHECK(L_out > 0, "Calculated output length is non-positive.");

    // Create output tensor
    auto y = torch::empty({N, C_out, L_out}, x.options().dtype(at::kFloat));

    // Bias pointer (may be null if bias is not provided)
    const float* bias_ptr = nullptr;
    if (bias_opt.has_value() && bias_opt.value().defined()) {
        TORCH_CHECK(bias_opt.value().is_cuda(), "bias must be a CUDA tensor if provided");
        TORCH_CHECK(bias_opt.value().scalar_type() == at::kFloat, "bias must be float32");
        bias_ptr = bias_opt.value().data_ptr<float>();
    }

    // Launch optimized kernel
    int total_threads = N * C_out * L_out;
    int blockSize = 256;
    int gridSize  = (total_threads + blockSize - 1) / blockSize;

    conv1d_forward_kernel_opt<<<gridSize, blockSize>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        y.data_ptr<float>(),
        N, (int)C_in, (int)L_in, (int)C_out, (int)K,
        (int)stride, (int)padding, (int)dilation, (int)groups,
        (int)L_out
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "conv1d_forward_kernel_opt failed: ", hipGetErrorString(err));

    return y;
}

// ---------------------------------------------------------------------
// Pybind11 binding with optional bias under the same module name
// ---------------------------------------------------------------------
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "forward",
        [](at::Tensor x,
           at::Tensor weight,
           py::object bias_obj,
           int64_t stride,
           int64_t padding,
           int64_t dilation,
           int64_t groups) {
            c10::optional<at::Tensor> bias;
            if (!bias_obj.is_none()) {
                bias = bias_obj.cast<at::Tensor>();
            }
            return conv1d_forward_impl(x, weight, bias, stride, padding, dilation, groups);
        },
        "Optimized 1D Convolution forward (CUDA) with optional bias using __ldg() for aligned global memory access"
    );
}
