#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32
#define BLOCK_SIZE 256
#define FULL_MASK 0xffffffff

template<typename scalar_t>
__device__ __forceinline__ scalar_t warpReduceSum(scalar_t val) {
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(FULL_MASK, val, offset);
    }
    return val;
}

__global__ void conv3d_warp_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int out_channels,
    const int in_depth,
    const int in_height,
    const int in_width,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int out_depth,
    const int out_height,
    const int out_width,
    const int stride,
    const int padding,
    const int dilation,
    const int groups) {

    const int warp_id = threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;
    const int warps_per_block = BLOCK_SIZE / WARP_SIZE;
    const int num_warps = warps_per_block * gridDim.x;
    
    const int total_elements = batch_size * out_channels * out_depth * out_height * out_width;
    
    for (int idx = blockIdx.x * warps_per_block + warp_id; idx < total_elements; idx += num_warps) {
        const int w_out = idx % out_width;
        int tmp = idx / out_width;
        const int h_out = tmp % out_height;
        tmp /= out_height;
        const int d_out = tmp % out_depth;
        tmp /= out_depth;
        const int c_out = tmp % out_channels;
        const int b = tmp / out_channels;

        const int group = c_out / (out_channels / groups);
        const int in_channels_per_group = in_channels / groups;
        
        float sum = 0.0f;
        
        // Distribute kernel elements across warp lanes
        for (int ic = 0; ic < in_channels_per_group; ic++) {
            const int in_c = group * in_channels_per_group + ic;
            
            for (int k_idx = lane_id; k_idx < kernel_d * kernel_h * kernel_w; k_idx += WARP_SIZE) {
                const int kd = k_idx / (kernel_h * kernel_w);
                const int kh = (k_idx / kernel_w) % kernel_h;
                const int kw = k_idx % kernel_w;
                
                const int d_in = d_out * stride - padding + kd * dilation;
                const int h_in = h_out * stride - padding + kh * dilation;
                const int w_in = w_out * stride - padding + kw * dilation;
                
                if (d_in >= 0 && d_in < in_depth && 
                    h_in >= 0 && h_in < in_height && 
                    w_in >= 0 && w_in < in_width) {
                    
                    const int input_idx = ((b * in_channels + in_c) * in_depth + d_in) * 
                                        in_height * in_width + h_in * in_width + w_in;
                    const int weight_idx = ((c_out * in_channels_per_group + ic) * kernel_d + kd) * 
                                         kernel_h * kernel_w + kh * kernel_w + kw;
                    
                    sum += input[input_idx] * weight[weight_idx];
                }
            }
        }
        
        // Warp-level reduction
        sum = warpReduceSum(sum);
        
        // First lane writes the result
        if (lane_id == 0) {
            if (bias != nullptr) {
                sum += bias[c_out];
            }
            output[idx] = sum;
        }
    }
}

at::Tensor forward(
    const at::Tensor& input,
    const at::Tensor& weight,
    const c10::optional<at::Tensor>& bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups) {
    
    auto bias = bias_opt.value_or(at::Tensor());
    
    TORCH_CHECK(input.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "Weight must be a CUDA tensor");
    TORCH_CHECK(!bias.defined() || bias.is_cuda(), "Bias must be a CUDA tensor");
    
    auto batch_size = input.size(0);
    auto in_channels = input.size(1);
    auto in_depth = input.size(2);
    auto in_height = input.size(3);
    auto in_width = input.size(4);
    
    auto out_channels = weight.size(0);
    auto kernel_d = weight.size(2);
    auto kernel_h = weight.size(3);
    auto kernel_w = weight.size(4);
    
    auto out_depth = (in_depth + 2 * padding - dilation * (kernel_d - 1) - 1) / stride + 1;
    auto out_height = (in_height + 2 * padding - dilation * (kernel_h - 1) - 1) / stride + 1;
    auto out_width = (in_width + 2 * padding - dilation * (kernel_w - 1) - 1) / stride + 1;
    
    auto output = at::empty({batch_size, out_channels, out_depth, out_height, out_width}, input.options());
    
    const int total_elements = batch_size * out_channels * out_depth * out_height * out_width;
    const int num_blocks = (total_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    conv3d_warp_kernel<<<num_blocks, BLOCK_SIZE>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.defined() ? bias.data_ptr<float>() : nullptr,
        output.data_ptr<float>(),
        batch_size, in_channels, out_channels,
        in_depth, in_height, in_width,
        kernel_d, kernel_h, kernel_w,
        out_depth, out_height, out_width,
        stride, padding, dilation, groups
    );
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D convolution forward with warp primitives (CUDA)");
}