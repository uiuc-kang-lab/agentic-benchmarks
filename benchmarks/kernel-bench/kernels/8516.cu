#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

namespace py = pybind11;

inline std::vector<int64_t> parseIntArrayRef(const py::object& obj) {
    std::vector<int64_t> result;
    if (py::isinstance<py::int_>(obj)) {
        result.push_back(obj.cast<int64_t>());
    } else if (py::isinstance<py::sequence>(obj)) {
        for (auto item : obj.cast<py::sequence>()) {
            result.push_back(py::cast<int64_t>(item));
        }
    } else {
        throw std::runtime_error("Expected int or sequence of ints");
    }
    return result;
}

__global__ void conv_transpose2d_blocksize32_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int h_in,
    const int w_in,
    const int out_channels,
    const int h_out,
    const int w_out,
    const int kernel_size,
    const int stride_h,
    const int stride_w,
    const int padding_h,
    const int padding_w,
    const int groups,
    const int in_channels_per_group,
    const int out_channels_per_group
) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int total_threads = gridDim.x * blockDim.x;
    const int thread_id = bid * blockDim.x + tid;
    
    const int total_elements = batch_size * out_channels * h_out * w_out;
    for (int index = thread_id; index < total_elements; index += total_threads) {
        const int w = index % w_out;
        int tmp = index / w_out;
        const int h = tmp % h_out;
        tmp = tmp / h_out;
        const int c = tmp % out_channels;
        const int n = tmp / out_channels;

        const int g = c / out_channels_per_group;
        const int c_local = c % out_channels_per_group;

        float sum = 0.0f;
        
        #pragma unroll 4
        for (int kh = 0; kh < kernel_size; ++kh) {
            #pragma unroll 4
            for (int kw = 0; kw < kernel_size; ++kw) {
                const int h_in_candidate = h + padding_h - kh;
                const int w_in_candidate = w + padding_w - kw;
                
                if ((h_in_candidate % stride_h == 0) && (w_in_candidate % stride_w == 0)) {
                    const int h_in_idx = h_in_candidate / stride_h;
                    const int w_in_idx = w_in_candidate / stride_w;
                    
                    if (h_in_idx >= 0 && h_in_idx < h_in && w_in_idx >= 0 && w_in_idx < w_in) {
                        #pragma unroll 4
                        for (int r = 0; r < in_channels_per_group; ++r) {
                            const int in_channel = g * in_channels_per_group + r;
                            const int input_idx = ((n * in_channels + in_channel) * h_in + h_in_idx) * w_in + w_in_idx;
                            const int weight_idx = (((g * in_channels_per_group + r) * out_channels_per_group + c_local) * kernel_size + kh) * kernel_size + kw;
                            
                            sum += __ldg(&input[input_idx]) * __ldg(&weight[weight_idx]);
                        }
                    }
                }
            }
        }

        if (bias != nullptr) {
            sum += __ldg(&bias[c]);
        }

        const int output_idx = ((n * out_channels + c) * h_out + h) * w_out + w;
        output[output_idx] = sum;
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    py::object stride = py::int_(1),
    py::object padding = py::int_(0),
    py::object output_padding = py::int_(0),
    int64_t groups = 1
) {
    auto stride_vec = parseIntArrayRef(stride);
    auto padding_vec = parseIntArrayRef(padding);
    auto output_padding_vec = parseIntArrayRef(output_padding);

    const int stride_h = stride_vec[0];
    const int stride_w = (stride_vec.size() > 1) ? stride_vec[1] : stride_h;
    const int padding_h = padding_vec[0];
    const int padding_w = (padding_vec.size() > 1) ? padding_vec[1] : padding_h;
    const int output_padding_h = output_padding_vec[0];
    const int output_padding_w = (output_padding_vec.size() > 1) ? output_padding_vec[1] : output_padding_h;

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int h_in = x.size(2);
    const int w_in = x.size(3);
    const int kernel_size = weight.size(2);
    const int out_channels = weight.size(1) * groups;

    const int h_out = (h_in - 1) * stride_h - 2 * padding_h + kernel_size + output_padding_h;
    const int w_out = (w_in - 1) * stride_w - 2 * padding_w + kernel_size + output_padding_w;

    auto output_tensor = torch::zeros({batch_size, out_channels, h_out, w_out}, x.options());

    const int in_channels_per_group = in_channels / groups;
    const int out_channels_per_group = out_channels / groups;

    const int block_size = 32;
    const int total_elements = batch_size * out_channels * h_out * w_out;
    const int num_blocks = (total_elements + block_size - 1) / block_size;
    const int max_blocks = 65535;
    const int grid_size = min(num_blocks, max_blocks);

    const float* input_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float* output_ptr = output_tensor.data_ptr<float>();

    conv_transpose2d_blocksize32_kernel<<<grid_size, block_size>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch_size,
        in_channels,
        h_in,
        w_in,
        out_channels,
        h_out,
        w_out,
        kernel_size,
        stride_h,
        stride_w,
        padding_h,
        padding_w,
        groups,
        in_channels_per_group,
        out_channels_per_group
    );

    hipDeviceSynchronize();
    return output_tensor;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ConvTranspose2d forward with block size 32",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride") = 1,
          py::arg("padding") = 0,
          py::arg("output_padding") = 0,
          py::arg("groups") = 1);
}