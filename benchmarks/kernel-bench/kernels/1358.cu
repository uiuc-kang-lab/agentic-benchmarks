#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel: Each block processes one row of B. The diagonal element from A is loaded once into shared memory.
__global__ void diag_matmul_shared_coalesced_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    const int64_t N,
    const int64_t M
) {
    int row = blockIdx.x; // Each block handles one row
    __shared__ float a_val;
    
    // Load the diagonal element for the current row into shared memory
    if (threadIdx.x == 0) {
        a_val = A[row];
    }
    __syncthreads(); // Synchronize to ensure a_val is available to all threads in the block
    
    // Compute output for the given row with coalesced memory access
    for (int col = threadIdx.x; col < M; col += blockDim.x) {
        // Aligned access for coalesced memory reading
        float b_val = __ldg(&B[row * M + col]);
        C[row * M + col] = a_val * b_val;
    }
}

// Forward function wraps the CUDA kernel
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.dim() == 1, "A must be a 1D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == B.size(0),
                "Dimension mismatch: A.size(0) must match B.size(0)");

    // Ensure inputs are contiguous
    A = A.contiguous();
    B = B.contiguous();

    int64_t N = A.size(0);
    int64_t M = B.size(1);

    // Create output tensor with the same type and device as B
    auto C = torch::empty({N, M}, B.options());

    // Launch one block per row with a fixed number of threads per block
    const int threads = 256;
    diag_matmul_shared_coalesced_kernel<<<N, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N,
        M
    );

    return C;
}

// Pybind11 module definition
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Diagonal matrix multiplication with coalesced memory access");
}