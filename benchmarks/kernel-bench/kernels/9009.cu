#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device function that performs the convolution for one output element.
__device__ __forceinline__ float compute_conv1d(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    int b,
    int oc,
    int o,
    int in_channels,
    int in_size,
    int kernel_size,
    int stride,
    int dilation) {
  float sum = 0.0f;
  int start_pos = o * stride;
  int end_pos = start_pos + (kernel_size - 1) * dilation;

  // If the convolution window is fully in bounds, avoid per-element boundary checks
  if (end_pos < in_size) {
    for (int ic = 0; ic < in_channels; ++ic) {
      const float* x_ptr = x + b * (in_channels * in_size) + ic * in_size + start_pos;
      const float* w_ptr = weight + oc * in_channels * kernel_size + ic * kernel_size;
      #pragma unroll
      for (int k = 0; k < kernel_size; ++k) {
        sum += x_ptr[k * dilation] * w_ptr[k];
      }
    }
  } else {
    // In boundary cases, check each position
    for (int ic = 0; ic < in_channels; ++ic) {
      const float* x_ptr = x + b * (in_channels * in_size) + ic * in_size;
      const float* w_ptr = weight + oc * in_channels * kernel_size + ic * kernel_size;
      #pragma unroll
      for (int k = 0; k < kernel_size; ++k) {
        int pos = start_pos + k * dilation;
        if (pos < in_size) {
          sum += x_ptr[pos] * w_ptr[k];
        }
      }
    }
  }
  return sum;
}

// CUDA kernel that computes the 1D convolution using shared memory for weights
__global__ void conv1d_shared_memory_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int B,
    int in_channels,
    int in_size,
    int out_channels,
    int kernel_size,
    int out_size,
    int stride,
    int dilation) {
  extern __shared__ float shared_weight[];

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total_elements = B * out_channels * out_size;
  if (idx >= total_elements) return;

  // Decode the linear index into (b, oc, o)
  int o = idx % out_size;
  int tmp = idx / out_size;
  int oc = tmp % out_channels;
  int b = tmp / out_channels;

  // Load weights into shared memory
  for (int ic = threadIdx.x; ic < in_channels * kernel_size; ic += blockDim.x) {
    shared_weight[ic] = weight[oc * in_channels * kernel_size + ic];
  }
  __syncthreads();

  // Compute the convolution using the modular device function
  float sum = compute_conv1d(x, shared_weight, b, oc, o, in_channels, in_size, kernel_size, stride, dilation);

  // Add bias if provided
  if (bias != nullptr) {
    sum += bias[oc];
  }

  int out_idx = b * (out_channels * out_size) + oc * out_size + o;
  output[out_idx] = sum;
}

// Forward function exposed via pybind11
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int dilation) {
  TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
  TORCH_CHECK(weight.device().is_cuda(), "weight must be a CUDA tensor");
  TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
  TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");
  TORCH_CHECK(x.dim() == 3, "x must be 3D");
  TORCH_CHECK(weight.dim() == 3, "weight must be 3D");
  TORCH_CHECK(weight.size(1) == x.size(1), "Input channels mismatch");

  if (bias.has_value()) {
    TORCH_CHECK(bias.value().device().is_cuda(), "bias must be a CUDA tensor");
    TORCH_CHECK(bias.value().is_contiguous(), "bias must be contiguous");
    TORCH_CHECK(bias.value().dim() == 1, "bias must be 1D");
    TORCH_CHECK(bias.value().size(0) == weight.size(0), "Bias size mismatch");
  }

  int B = x.size(0);
  int in_channels = x.size(1);
  int in_size = x.size(2);
  int out_channels = weight.size(0);
  int kernel_size = weight.size(2);

  int out_size = (in_size - dilation * (kernel_size - 1) - 1) / stride + 1;
  TORCH_CHECK(out_size > 0, "Invalid output size");

  auto output = torch::empty({B, out_channels, out_size}, x.options());
  if (output.numel() == 0) return output;

  const float* x_data = x.data_ptr<float>();
  const float* weight_data = weight.data_ptr<float>();
  const float* bias_data = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
  float* output_data = output.data_ptr<float>();

  int total_elements = B * out_channels * out_size;
  int threads = 256;
  int blocks = (total_elements + threads - 1) / threads;

  size_t shared_memory_size = in_channels * kernel_size * sizeof(float);

  conv1d_shared_memory_kernel<<<blocks, threads, shared_memory_size>>>(
      x_data,
      weight_data,
      bias_data,
      output_data,
      B,
      in_channels,
      in_size,
      out_channels,
      kernel_size,
      out_size,
      stride,
      dilation);

  hipError_t err = hipGetLastError();
  TORCH_CHECK(err == hipSuccess, "CUDA kernel launch error: ", hipGetErrorString(err));

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "Modular 1D convolution forward with shared memory (CUDA)");
}
