#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static const int BLOCK_SIZE = 256;

template <typename scalar_t>
__global__ void mse_forward_kernel_optimized(
    const scalar_t* __restrict__ preds,
    const scalar_t* __restrict__ tgts,
    double* __restrict__ sum_out,
    const int64_t num_elements
) {
    __shared__ double shm[BLOCK_SIZE];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double thread_sum = 0.0;

    // Strided loop to accumulate squared differences
    while (idx < num_elements) {
        double diff = static_cast<double>(preds[idx]) - static_cast<double>(tgts[idx]);
        thread_sum += diff * diff;
        idx += blockDim.x * gridDim.x;
    }

    // Store partial sums in shared memory
    shm[threadIdx.x] = thread_sum;
    __syncthreads();

    // Block-level reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shm[threadIdx.x] += shm[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Only one thread per block performs atomic addition to global memory
    if (threadIdx.x == 0) {
        atomicAdd(sum_out, shm[0]);
    }
}

torch::Tensor forward_optimized(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.is_cuda(), "predictions must be a CUDA tensor");
    TORCH_CHECK(targets.is_cuda(), "targets must be a CUDA tensor");
    TORCH_CHECK(predictions.numel() == targets.numel(),
                "predictions and targets must have the same number of elements");

    const int64_t num_elements = predictions.numel();
    // Use double for accumulation
    auto accumulator = torch::zeros({1}, predictions.options().dtype(at::kDouble));

    const int grid_size = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    AT_DISPATCH_FLOATING_TYPES(predictions.scalar_type(), "mse_forward_cuda_optimized", [&] {
        mse_forward_kernel_optimized<scalar_t><<<grid_size, BLOCK_SIZE>>>(
            predictions.data_ptr<scalar_t>(),
            targets.data_ptr<scalar_t>(),
            accumulator.data_ptr<double>(),
            num_elements
        );
    });

    // Final mean = accumulator / N
    auto result = accumulator.div_(static_cast<double>(num_elements));
    return result.to(predictions.dtype());
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward_optimized", &forward_optimized, "Mean Squared Error (MSE) forward optimized (CUDA)");
}