#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 16
#define CHANNELS_PER_BLOCK 8

__global__ void conv2d_kernel_tuned(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int input_height,
    const int input_width,
    const int out_channels,
    const int kernel_h,
    const int kernel_w,
    const int height_out,
    const int width_out,
    const int stride,
    const int pad_h,
    const int pad_w,
    const int dilation_h,
    const int dilation_w) {

    

    const int w_out = blockIdx.x * BLOCK_SIZE_X + threadIdx.x;
    const int h_out = blockIdx.y * BLOCK_SIZE_Y + threadIdx.y;
    const int oc_base = blockIdx.z * CHANNELS_PER_BLOCK;

    if (h_out >= height_out || w_out >= width_out) return;

    float sums[CHANNELS_PER_BLOCK] = {0.0f};
    
    #pragma unroll
    for (int oc_offset = 0; oc_offset < CHANNELS_PER_BLOCK && oc_base + oc_offset < out_channels; ++oc_offset) {
        sums[oc_offset] = bias ? bias[oc_base + oc_offset] : 0.0f;
    }

    for (int b = 0; b < batch_size; ++b) {
        for (int ic = 0; ic < in_channels; ++ic) {
            #pragma unroll 4
            for (int kh = 0; kh < kernel_h; ++kh) {
                const int h_in = h_out * stride + kh * dilation_h - pad_h;
                
                if (h_in >= 0 && h_in < input_height) {
                    #pragma unroll 4
                    for (int kw = 0; kw < kernel_w; ++kw) {
                        const int w_in = w_out * stride + kw * dilation_w - pad_w;
                        
                        if (w_in >= 0 && w_in < input_width) {
                            const float x_val = x[b * in_channels * input_height * input_width +
                                               ic * input_height * input_width +
                                               h_in * input_width + w_in];
                            
                            #pragma unroll
                            for (int oc_offset = 0; oc_offset < CHANNELS_PER_BLOCK && oc_base + oc_offset < out_channels; ++oc_offset) {
                                const float w_val = weight[(oc_base + oc_offset) * in_channels * kernel_h * kernel_w +
                                                         ic * kernel_h * kernel_w +
                                                         kh * kernel_w + kw];
                                sums[oc_offset] = __fmaf_rn(x_val, w_val, sums[oc_offset]);
                            }
                        }
                    }
                }
            }
        }

        #pragma unroll
        for (int oc_offset = 0; oc_offset < CHANNELS_PER_BLOCK && oc_base + oc_offset < out_channels; ++oc_offset) {
            const int out_idx = b * out_channels * height_out * width_out +
                               (oc_base + oc_offset) * height_out * width_out +
                               h_out * width_out + w_out;
            output[out_idx] = sums[oc_offset];
            sums[oc_offset] = bias ? bias[oc_base + oc_offset] : 0.0f;
        }
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    std::tuple<int, int> padding,
    std::tuple<int, int> dilation) {

    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");

    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        TORCH_CHECK(bias->is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        bias_ptr = bias->data_ptr<float>();
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int input_height = x.size(2);
    int input_width = x.size(3);
    int out_channels = weight.size(0);
    int kernel_h = weight.size(2);
    int kernel_w = weight.size(3);
    int pad_h = std::get<0>(padding);
    int pad_w = std::get<1>(padding);
    int dilation_h = std::get<0>(dilation);
    int dilation_w = std::get<1>(dilation);

    int height_out = (input_height + 2 * pad_h - dilation_h * (kernel_h - 1) - 1) / stride + 1;
    int width_out = (input_width + 2 * pad_w - dilation_w * (kernel_w - 1) - 1) / stride + 1;

    auto output = torch::empty({batch_size, out_channels, height_out, width_out}, x.options());

    if (height_out == 0 || width_out == 0) return output;

    dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 blocks(
        (width_out + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X,
        (height_out + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y,
        (out_channels + CHANNELS_PER_BLOCK - 1) / CHANNELS_PER_BLOCK
    );

    conv2d_kernel_tuned<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        input_height,
        input_width,
        out_channels,
        kernel_h,
        kernel_w,
        height_out,
        width_out,
        stride,
        pad_h,
        pad_w,
        dilation_h,
        dilation_w
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Conv2D forward (CUDA)");
}