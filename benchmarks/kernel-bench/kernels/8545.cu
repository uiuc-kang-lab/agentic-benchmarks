#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

namespace py = pybind11;

// Helper to parse int or sequence of ints
inline std::vector<int64_t> parseIntArrayRef(const py::object &obj) {
    std::vector<int64_t> result;
    if (py::isinstance<py::int_>(obj)) {
        // If single integer is provided, duplicate it for both dimensions
        int64_t val = obj.cast<int64_t>();
        result.push_back(val);
        result.push_back(val);
    } else if (py::isinstance<py::sequence>(obj)) {
        for (auto item : obj.cast<py::sequence>()) {
            result.push_back(py::cast<int64_t>(item));
        }
        // If sequence has only one element, duplicate it
        if (result.size() == 1) {
            result.push_back(result[0]);
        }
    } else {
        throw std::runtime_error("Expected int or sequence of ints");
    }
    
    if (result.size() != 2) {
        throw std::runtime_error("Must provide either a single integer or 2 integers for 2D operation");
    }
    return result;
}

// CUDA kernel for transposed 2D convolution ensuring coalesced global memory accesses
__global__ void conv_transposed2d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_h,
    int in_w,
    int out_h,
    int out_w,
    int kernel_h,
    int kernel_w,
    int stride_h,
    int stride_w,
    int pad_h,
    int pad_w,
    int in_channels_per_group,
    int out_channels_per_group
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch_size * out_channels * out_h * out_w;
    
    // Each thread computes one output element. With the flattened index, writes are coalesced.
    while (idx < total) {
        int ow = idx % out_w;
        int temp = idx / out_w;
        int oh = temp % out_h;
        temp /= out_h;
        int oc = temp % out_channels;
        int n = temp / out_channels;
        
        float sum = 0.0f;

        // Determine the corresponding group
        int group = oc / out_channels_per_group;
        int start_ic = group * in_channels_per_group;
        int end_ic = start_ic + in_channels_per_group;

        // For every input channel in the corresponding group
        for (int ic = start_ic; ic < end_ic; ic++) {
            // Loop over the kernel spatial dimensions
            for (int kh = 0; kh < kernel_h; kh++) {
                int i_h = oh + pad_h - kh;
                if (i_h % stride_h != 0) continue;  // Not a valid contributing position
                int i_h_div = i_h / stride_h;
                if (i_h_div < 0 || i_h_div >= in_h) continue;
                
                for (int kw = 0; kw < kernel_w; kw++) {
                    int i_w = ow + pad_w - kw;
                    if (i_w % stride_w != 0) continue;
                    int i_w_div = i_w / stride_w;
                    if (i_w_div < 0 || i_w_div >= in_w) continue;

                    // Compute input and weight indices
                    int input_index = ((n * in_channels + ic) * in_h + i_h_div) * in_w + i_w_div;
                    int weight_index = (ic * out_channels_per_group + (oc % out_channels_per_group)) * (kernel_h * kernel_w) + kh * kernel_w + kw;

                    sum += input[input_index] * weight[weight_index];
                }
            }
        }

        // Add bias if provided
        if (bias != nullptr) {
            sum += bias[oc];
        }
        
        output[idx] = sum;
        idx += blockDim.x * gridDim.x;
    }
}

// Forward function wrapper for the CUDA kernel
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    py::object stride = py::int_(1),
    py::object padding = py::int_(0),
    py::object output_padding = py::int_(0),
    int64_t groups = 1
) {
    // Parse stride, padding, and output_padding. Expect 2-element sequences for 2D ops.
    auto stride_vec = parseIntArrayRef(stride);
    auto padding_vec = parseIntArrayRef(padding);
    auto output_padding_vec = parseIntArrayRef(output_padding);
    if (stride_vec.size() != 2 || padding_vec.size() != 2 || output_padding_vec.size() != 2) {
        throw std::runtime_error("stride, padding, and output_padding must be 2-element sequences");
    }
    int stride_h = stride_vec[0];
    int stride_w = stride_vec[1];
    int pad_h = padding_vec[0];
    int pad_w = padding_vec[1];
    int output_pad_h = output_padding_vec[0];
    int output_pad_w = output_padding_vec[1];

    // Input dimensions (N, C, H, W)
    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int in_h = x.size(2);
    int in_w = x.size(3);

    // Weight dimensions: assumed to be [in_channels, out_channels/group, kernel_h, kernel_w]
    int kernel_h = weight.size(2);
    int kernel_w = weight.size(3);
    int out_channels_per_group = weight.size(1);
    int out_channels = out_channels_per_group * groups;

    // Compute output spatial dimensions for conv_transpose2d
    int out_h = (in_h - 1) * stride_h - 2 * pad_h + kernel_h + output_pad_h;
    int out_w = (in_w - 1) * stride_w - 2 * pad_w + kernel_w + output_pad_w;

    auto output = torch::zeros({batch_size, out_channels, out_h, out_w}, x.options());

    // Compute channels per group
    int in_channels_per_group = in_channels / groups;

    int total = batch_size * out_channels * out_h * out_w;
    int threads = 256;
    int blocks = (total + threads - 1) / threads;

    x = x.contiguous();
    weight = weight.contiguous();
    torch::Tensor bias_tensor;
    if (bias.has_value() && bias.value().defined()) {
        bias_tensor = bias.value().contiguous();
    }

    const float* x_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = (bias_tensor.defined()) ? bias_tensor.data_ptr<float>() : nullptr;
    float* output_ptr = output.data_ptr<float>();

    conv_transposed2d_kernel<<<blocks, threads>>>(
        x_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch_size,
        in_channels,
        out_channels,
        in_h,
        in_w,
        out_h,
        out_w,
        kernel_h,
        kernel_w,
        stride_h,
        stride_w,
        pad_h,
        pad_w,
        in_channels_per_group,
        out_channels_per_group
    );
    hipDeviceSynchronize();

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ConvTranspose2d forward with coalesced global memory access",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride") = 1,
          py::arg("padding") = 0,
          py::arg("output_padding") = 0,
          py::arg("groups") = 1);
}
