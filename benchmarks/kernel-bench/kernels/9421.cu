#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Allow configurable block size. Experiment with 32, 64, 128, 256, 512
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256
#endif

__global__ void conv2d_kernel_optimized(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int input_height,
    int input_width,
    int out_channels,
    int kernel_h,
    int kernel_w,
    int height_out,
    int width_out,
    int stride,
    int pad_h,
    int pad_w,
    int dilation_h,
    int dilation_w) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch_size * out_channels * height_out * width_out;
    if (idx >= total) return;

    int w_out = idx % width_out;
    int temp = idx / width_out;
    int h_out = temp % height_out;
    temp /= height_out;
    int oc = temp % out_channels;
    int b = temp / out_channels;

    float sum = bias ? bias[oc] : 0.0f;

    for (int ic = 0; ic < in_channels; ++ic) {
        for (int kh = 0; kh < kernel_h; ++kh) {
            int h_in = h_out * stride + kh * dilation_h - pad_h;
            if (h_in < 0 || h_in >= input_height) continue;
            for (int kw = 0; kw < kernel_w; ++kw) {
                int w_in = w_out * stride + kw * dilation_w - pad_w;
                if (w_in < 0 || w_in >= input_width) continue;
                int x_idx = b * in_channels * input_height * input_width
                          + ic * input_height * input_width
                          + h_in * input_width + w_in;
                int w_idx = oc * in_channels * kernel_h * kernel_w
                          + ic * kernel_h * kernel_w
                          + kh * kernel_w + kw;
                sum += x[x_idx] * weight[w_idx];
            }
        }
    }

    int out_idx = b * out_channels * height_out * width_out
                + oc * height_out * width_out
                + h_out * width_out + w_out;
    output[out_idx] = sum;
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,  // Optional bias
    int stride,
    std::tuple<int, int> padding,
    std::tuple<int, int> dilation) {

    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");

    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        TORCH_CHECK(bias->is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        bias_ptr = bias->data_ptr<float>();
    }

    int batch_size = x.size(0);
    int in_channels = x.size(1);
    int input_height = x.size(2);
    int input_width = x.size(3);

    int out_channels = weight.size(0);
    int kernel_h = weight.size(2);
    int kernel_w = weight.size(3);

    int pad_h = std::get<0>(padding);
    int pad_w = std::get<1>(padding);
    int dilation_h = std::get<0>(dilation);
    int dilation_w = std::get<1>(dilation);

    int height_out = (input_height + 2 * pad_h - dilation_h * (kernel_h - 1) - 1) / stride + 1;
    int width_out = (input_width + 2 * pad_w - dilation_w * (kernel_w - 1) - 1) / stride + 1;

    auto output = torch::empty({batch_size, out_channels, height_out, width_out}, x.options());

    int total_elements = batch_size * out_channels * height_out * width_out;
    if (total_elements <= 0) return output;

    int threads = BLOCK_SIZE;
    int blocks = (total_elements + threads - 1) / threads;

    conv2d_kernel_optimized<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        input_height,
        input_width,
        out_channels,
        kernel_h,
        kernel_w,
        height_out,
        width_out,
        stride,
        pad_h,
        pad_w,
        dilation_h,
        dilation_w);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Conv2D forward optimized with block size tuning (CUDA)");
}
