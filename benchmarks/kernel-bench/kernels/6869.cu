#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <float.h>

__global__ void argmax_kernel_warp_prim(
    const float* __restrict__ x,
    int64_t* __restrict__ indices,
    const int outerSize,
    const int dimSize,
    const int innerSize) {

    int slice = blockIdx.x;
    if (slice >= outerSize * innerSize) return;

    int outer_idx = slice / innerSize;
    int inner_idx = slice % innerSize;
    int base_offset = outer_idx * (dimSize * innerSize) + inner_idx;

    float local_max = -FLT_MAX;
    int local_argmax = 0;

    for (int d = threadIdx.x; d < dimSize; d += blockDim.x) {
        float curr_val = x[base_offset + d * innerSize];
        if (curr_val > local_max) {
            local_max = curr_val;
            local_argmax = d;
        }
    }

    // Use warp-level primitives for reduction
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        float max_shuffled = __shfl_down_sync(0xffffffff, local_max, offset);
        int idx_shuffled = __shfl_down_sync(0xffffffff, local_argmax, offset);
        if (max_shuffled > local_max) {
            local_max = max_shuffled;
            local_argmax = idx_shuffled;
        }
    }

    if (threadIdx.x % warpSize == 0) {
        atomicMax(&indices[slice], local_argmax);
    }
}

torch::Tensor argmax_forward_cuda(const torch::Tensor& x, const int64_t dim) {
    TORCH_CHECK(x.scalar_type() == at::kFloat, "Only float32 is supported.");
    auto x_contig = x.contiguous();

    auto sizes = x_contig.sizes();
    int ndim = x_contig.dim();
    TORCH_CHECK(dim >= 0 && dim < ndim, "Invalid dim for argmax.");

    int outerSize = 1;
    for (int d = 0; d < dim; d++) {
        outerSize *= sizes[d];
    }
    int dimSize = sizes[dim];
    int innerSize = 1;
    for (int d = dim + 1; d < ndim; d++) {
        innerSize *= sizes[d];
    }

    std::vector<int64_t> out_sizes;
    for (int d = 0; d < ndim; d++) {
        if (d == dim) continue;
        out_sizes.push_back(sizes[d]);
    }
    auto options = torch::TensorOptions().device(x.device()).dtype(torch::kLong);
    auto indices = torch::empty(out_sizes, options);

    int slices = outerSize * innerSize;

    const int threads = 128;
    int blocks = slices;

    argmax_kernel_warp_prim<<<blocks, threads>>>(
        x_contig.data_ptr<float>(),
        indices.data_ptr<int64_t>(),
        outerSize,
        dimSize,
        innerSize
    );

    return indices;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &argmax_forward_cuda, "ArgMax CUDA forward with warp primitives");
}
