#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32
#define VECTOR_SIZE 4  // Process 4 elements at once using float4

__device__ __forceinline__ void load_tile_vectorized(const float* __restrict__ src,
                                                    float dst[TILE_SIZE][TILE_SIZE+1],
                                                    const int row, const int col,
                                                    const int N, const int stride) {
    float4 vec;
    int base_idx = row * stride + col;
    
    if (col + 4 <= N && row < N) {
        vec = *reinterpret_cast<const float4*>(&src[base_idx]);
        dst[threadIdx.y][threadIdx.x * 4] = vec.x;
        dst[threadIdx.y][threadIdx.x * 4 + 1] = vec.y;
        dst[threadIdx.y][threadIdx.x * 4 + 2] = vec.z;
        dst[threadIdx.y][threadIdx.x * 4 + 3] = vec.w;
    } else {
        for (int i = 0; i < 4; i++) {
            if (col + i < N && row < N) {
                dst[threadIdx.y][threadIdx.x * 4 + i] = src[base_idx + i];
            } else {
                dst[threadIdx.y][threadIdx.x * 4 + i] = 0.0f;
            }
        }
    }
}

__global__ void vectorized_triangular_mm_kernel(const float* __restrict__ A,
                                              const float* __restrict__ B,
                                              float* __restrict__ C,
                                              const int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE+1];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE+1];

    // Each thread now handles 4 elements
    const int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    const int col_base = blockIdx.x * TILE_SIZE + threadIdx.x * 4;

    float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    
    // Early exit if entire vector is in upper triangle
    if (row < (col_base)) {
        if (row < N && col_base < N) {
            for (int i = 0; i < 4; i++) {
                if (col_base + i < N) {
                    C[row * N + col_base + i] = 0.0f;
                }
            }
        }
        return;
    }

    const int num_tiles = (N + TILE_SIZE - 1) / TILE_SIZE;
    
    for (int t = 0; t < num_tiles; t++) {
        const int tile_start = t * TILE_SIZE;
        if (tile_start > row) break;

        // Load tiles using vectorized loads where possible
        load_tile_vectorized(A, As, row, tile_start + threadIdx.x * 4, N, N);
        load_tile_vectorized(B, Bs, tile_start + threadIdx.y, col_base, N, N);
        
        __syncthreads();

        const int k_start = max(tile_start, col_base);
        const int k_end = min(tile_start + TILE_SIZE, row + 1);

        #pragma unroll 8
        for (int k = k_start; k < k_end; k++) {
            const float a_val = As[threadIdx.y][k - tile_start];
            const int bs_idx = k - tile_start;
            
            sum.x += a_val * Bs[bs_idx][threadIdx.x * 4];
            sum.y += a_val * Bs[bs_idx][threadIdx.x * 4 + 1];
            sum.z += a_val * Bs[bs_idx][threadIdx.x * 4 + 2];
            sum.w += a_val * Bs[bs_idx][threadIdx.x * 4 + 3];
        }
        
        __syncthreads();
    }

    // Write results back to global memory
    if (row < N) {
        for (int i = 0; i < 4; i++) {
            const int col = col_base + i;
            if (col < N) {
                if (row >= col) {
                    float result;
                    switch(i) {
                        case 0: result = sum.x; break;
                        case 1: result = sum.y; break;
                        case 2: result = sum.z; break;
                        case 3: result = sum.w; break;
                    }
                    C[row * N + col] = result;
                } else {
                    C[row * N + col] = 0.0f;
                }
            }
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 block(TILE_SIZE/4, TILE_SIZE);  // Adjust block size for vectorization
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE,
              (N + TILE_SIZE - 1) / TILE_SIZE);

    vectorized_triangular_mm_kernel<<<grid, block>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Vectorized lower triangular matrix multiplication (CUDA)");
}