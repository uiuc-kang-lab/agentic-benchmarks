#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void triangular_mm_kernel_stride_optimized(const float* __restrict__ A,
                                                      const float* __restrict__ B,
                                                      float* __restrict__ C,
                                                      int N) {
    // Acquire global row and column indices using stride loops.
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    for (int stride_row = row; stride_row < N; stride_row += blockDim.y * gridDim.y) {
        for (int stride_col = col; stride_col <= stride_row && stride_col < N; stride_col += blockDim.x * gridDim.x) {
            float sum = 0.f;
            for (int k = stride_col; k <= stride_row; ++k) {
                sum += __ldg(&A[stride_row * N + k]) * __ldg(&B[k * N + stride_col]);
            }
            C[stride_row * N + stride_col] = sum;
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threads - 1) / threads, (N + threads - 1) / threads);

    triangular_mm_kernel_stride_optimized<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}
