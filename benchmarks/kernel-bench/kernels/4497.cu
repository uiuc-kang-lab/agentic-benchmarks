#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/cuda/HIPContext.h>
#include <vector>

typedef float4 float4_t;

// Optimized warp reduction with no divergent branches
template <typename T>
__device__ __forceinline__ T warpReduceSum(T val) {
    #pragma unroll
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

// Optimized block reduction with minimal synchronization
template <typename T>
__device__ __forceinline__ T blockReduceSum(T val) {
    static __shared__ T shared[32]; // Shared mem for 32 partial sums
    const int lid = threadIdx.x % warpSize;
    const int wid = threadIdx.x / warpSize;

    val = warpReduceSum(val); // First reduce within warps

    if (lid == 0) shared[wid] = val; // Write reduced warp values to shared mem
    
    __syncthreads(); // Single sync point - only one needed for reduction

    // First warp reduces final results
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lid] : 0;
    if (wid == 0) val = warpReduceSum(val);
    
    return val;
}

// Kernel to compute per-group mean and variance
// Each block is assigned one group for one batch element
template <typename scalar_t>
__global__ void compute_stats_kernel(
    const scalar_t* __restrict__ x,
    const int N,
    const int C,
    const int spatial,
    const int channels_per_group,
    const int num_groups,
    scalar_t* __restrict__ mean,
    scalar_t* __restrict__ var) {

    const int idx = blockIdx.x;
    const int n = idx / num_groups;
    const int g = idx % num_groups;
    
    const int group_offset = n * C * spatial + g * channels_per_group * spatial;
    const int group_elems = channels_per_group * spatial;
    
    const int vec_size = 4;
    const int num_vectors = group_elems / vec_size;
    const int remaining = group_elems % vec_size;
    
    scalar_t thread_sum = 0;
    scalar_t thread_sum_sq = 0;

    // Vectorized loads using __ldg
    const float4_t* x_vec = reinterpret_cast<const float4_t*>(x + group_offset);
    #pragma unroll 4
    for (int i = threadIdx.x; i < num_vectors; i += blockDim.x) {
        float4_t v = __ldg(x_vec + i);
        thread_sum += v.x + v.y + v.z + v.w;
        thread_sum_sq += v.x * v.x + v.y * v.y + v.z * v.z + v.w * v.w;
    }

    if (threadIdx.x < remaining) {
        const scalar_t val = __ldg(x + group_offset + num_vectors * vec_size + threadIdx.x);
        thread_sum += val;
        thread_sum_sq += val * val;
    }

    // Single block reduction call handles all reductions with minimal syncs
    thread_sum = blockReduceSum(thread_sum);
    thread_sum_sq = blockReduceSum(thread_sum_sq);

    if (threadIdx.x == 0) {
        const scalar_t group_mean = thread_sum / group_elems;
        const scalar_t group_var = thread_sum_sq / group_elems - group_mean * group_mean;
        const int out_index = n * num_groups + g;
        mean[out_index] = group_mean;
        var[out_index] = group_var;
    }
}

// Kernel to apply the group normalization
// Each thread processes one element from the input
template <typename scalar_t>
__global__ void group_norm_forward_kernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ mean,
    const scalar_t* __restrict__ var,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    const int N,
    const int C,
    const int spatial,
    const int channels_per_group,
    const int num_groups,
    const scalar_t eps,
    scalar_t* __restrict__ y) {

    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int total = N * C * spatial;

    // Process 4 elements per thread per iteration
    #pragma unroll 4
    for (int base_idx = tid * 4; base_idx < total; base_idx += stride * 4) {
        float4_t x_val = __ldg(reinterpret_cast<const float4_t*>(x + base_idx));
        float4_t result;

        #pragma unroll
        for (int i = 0; i < 4; i++) {
            const int idx = base_idx + i;
            if (idx < total) {
                const int j = idx % spatial;
                const int temp = idx / spatial;
                const int c = temp % C;
                const int n = temp / C;
                const int g = c / channels_per_group;
                const int stats_index = n * num_groups + g;

                const scalar_t m = __ldg(mean + stats_index);
                const scalar_t v = __ldg(var + stats_index);
                const scalar_t inv_std = rsqrt(v + eps);
                const scalar_t w = __ldg(weight + c);
                const scalar_t b = __ldg(bias + c);

                (&result.x)[i] = ((&x_val.x)[i] - m) * inv_std * w + b;
            }
        }
        
        *reinterpret_cast<float4_t*>(y + base_idx) = result;
    }
}

// Host function to launch the optimized kernels with CUDA streams
// This function uses multiple streams to overlap kernel execution with memory transfers
torch::Tensor group_norm_forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    int64_t num_groups,
    double eps) {

    const int N = x.size(0);
    const int C = x.size(1);
    int spatial = 1;
    for (int i = 2; i < x.dim(); i++) {
        spatial *= x.size(i);
    }
    const int channels_per_group = C / num_groups;

    auto y = torch::empty_like(x);
    auto options = torch::TensorOptions().device(x.device()).dtype(x.dtype());
    auto mean = torch::empty({N, num_groups}, options);
    auto var = torch::empty({N, num_groups}, options);

    const int total_groups = N * num_groups;
    const int threads_stats = 512; // Increased thread count for better occupancy
    const dim3 blocks_stats(total_groups);

    const int threads_norm = 256;
    const int blocks_norm = (N * C * spatial + threads_norm * 4 - 1) / (threads_norm * 4);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "group_norm_forward_cuda", ([&] {
        // Launch the kernel to compute means and variances on stream1
        compute_stats_kernel<scalar_t><<<blocks_stats, threads_stats, 0, stream1>>>(
            x.data_ptr<scalar_t>(),
            N, C, spatial,
            channels_per_group,
            num_groups,
            mean.data_ptr<scalar_t>(),
            var.data_ptr<scalar_t>());

        // Launch the kernel to perform group normalization on stream2
        group_norm_forward_kernel<scalar_t><<<blocks_norm, threads_norm, 0, stream2>>>(
            x.data_ptr<scalar_t>(),
            mean.data_ptr<scalar_t>(),
            var.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            bias.data_ptr<scalar_t>(),
            N, C, spatial,
            channels_per_group,
            num_groups,
            static_cast<scalar_t>(eps),
            y.data_ptr<scalar_t>());
    }));

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &group_norm_forward, "Group Normalization forward (CUDA) with pipelined streams");
}
