#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void avg_pool2d_forward_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int N, int C, int H, int W,
    int outH, int outW,
    int kernel_size, int stride, int padding
) {
    const unsigned int warp_size = 32;
    const unsigned int lane_id = threadIdx.x % warp_size;
    const unsigned int warp_id = threadIdx.x / warp_size;
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = N * C * outH * outW;
    if (index >= total) return;

    int w_out = index % outW;
    int h_out = (index / outW) % outH;
    int c = (index / (outW * outH)) % C;
    int n = index / (outW * outH * C);

    int h_start = h_out * stride - padding;
    int w_start = w_out * stride - padding;

    scalar_t sum_val = 0;
    const int elements_per_thread = (kernel_size * kernel_size + warp_size - 1) / warp_size;
    
    #pragma unroll
    for (int k = 0; k < elements_per_thread; k++) {
        int flat_idx = k * warp_size + lane_id;
        if (flat_idx < kernel_size * kernel_size) {
            int i = flat_idx / kernel_size;
            int j = flat_idx % kernel_size;
            
            int h_in = h_start + i;
            int w_in = w_start + j;
            
            if (h_in >= 0 && h_in < H && w_in >= 0 && w_in < W) {
                sum_val += input[((n * C + c) * H + h_in) * W + w_in];
            }
        }
    }

    // Warp-level reduction using shuffle operations
    #pragma unroll
    for (int offset = warp_size/2; offset > 0; offset >>= 1) {
        sum_val += __shfl_down_sync(0xffffffff, sum_val, offset);
    }

    if (lane_id == 0) {
        output[index] = sum_val / static_cast<scalar_t>(kernel_size * kernel_size);
    }
}

torch::Tensor avg_pool2d_forward(
    torch::Tensor x,
    int kernel_size,
    int stride,
    int padding
) {
    TORCH_CHECK(x.dim() == 4, "Input must be a 4D tensor.");
    auto N = x.size(0);
    auto C = x.size(1);
    auto H = x.size(2);
    auto W = x.size(3);

    int outH = (H + 2 * padding - kernel_size) / stride + 1;
    int outW = (W + 2 * padding - kernel_size) / stride + 1;

    auto x_cont = x.contiguous();
    auto options = x.options();
    auto out = torch::empty({N, C, outH, outW}, options);

    const int threads = 128;
    const int blocks = (N * C * outH * outW + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "avg_pool2d_forward_kernel", ([&] {
        const scalar_t* input_data = x_cont.data_ptr<scalar_t>();
        scalar_t* output_data = out.data_ptr<scalar_t>();

        avg_pool2d_forward_kernel<<<blocks, threads>>>(
            input_data,
            output_data,
            N, C, H, W,
            outH, outW,
            kernel_size, stride, padding
        );
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err));

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &avg_pool2d_forward, "2D Average Pooling forward (CUDA)");
}