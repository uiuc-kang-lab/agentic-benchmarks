#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// Device function to compute a valid input coordinate for transposed convolution.
__device__ inline bool get_valid_index(int out_coord, int pad, int k, int dilation, int stride, int input_dim, int &in_coord) {
    int unscaled = out_coord + pad - k * dilation;
    if (unscaled % stride != 0)
        return false;
    in_coord = unscaled / stride;
    return (in_coord >= 0 && in_coord < input_dim);
}

// CUDA kernel for 2D transposed convolution using modular device functions.
__global__ void conv_transpose2d_forward_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch_size * out_channels * out_height * out_width;
    if (index >= total)
        return;

    // Decode index into (b, o, out_h, out_w)
    int w_out = index % out_width;
    int temp = index / out_width;
    int h_out = temp % out_height;
    temp /= out_height;
    int o = temp % out_channels;
    int b = temp / out_channels;

    float out_val = bias[o];

    // Loop over input channels and kernel elements using the modular get_valid_index function
    for (int c = 0; c < in_channels; ++c) {
        for (int p = 0; p < kernel_size; ++p) {
            int h_in;
            if (!get_valid_index(h_out, padding, p, dilation, stride, in_height, h_in))
                continue;
            for (int q = 0; q < kernel_size; ++q) {
                int w_in;
                if (!get_valid_index(w_out, padding, q, dilation, stride, in_width, w_in))
                    continue;
                int input_idx = ((b * in_channels + c) * in_height + h_in) * in_width + w_in;
                int weight_idx = ((c * out_channels + o) * kernel_size + p) * kernel_size + q;
                out_val += input[input_idx] * weight[weight_idx];
            }
        }
    }

    int output_idx = ((b * out_channels + o) * out_height + h_out) * out_width + w_out;
    output[output_idx] = out_val;
}

// CUDA wrapper function to launch the kernel
torch::Tensor conv_transpose2d_forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {

    int batch_size = input.size(0);
    int in_channels = input.size(1);
    int in_height = input.size(2);
    int in_width = input.size(3);

    int out_channels = weight.size(1);
    int kernel_size = weight.size(2);  // assume square kernel

    int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
    int out_width  = (in_width - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;

    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());

    int total_threads = batch_size * out_channels * out_height * out_width;
    int threads = 1024;
    int blocks = (total_threads + threads - 1) / threads;

    conv_transpose2d_forward_kernel<<<blocks, threads>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        out_channels,
        in_height,
        in_width,
        kernel_size,
        out_height,
        out_width,
        stride,
        padding,
        dilation);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in conv_transpose2d_forward_kernel: %s\n", hipGetErrorString(err));
    }

    return output;
}

// Wrapper to handle the possibility of bias being None
torch::Tensor conv_transpose2d_forward_wrapper(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,
    int stride,
    int padding,
    int dilation) {

    int out_channels = weight.size(1);
    torch::Tensor bias;
    if (bias_obj.is(pybind11::none())) {
        bias = torch::zeros({out_channels}, weight.options());
    } else {
        bias = bias_obj.cast<torch::Tensor>();
    }

    return conv_transpose2d_forward_cuda(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv_transpose2d_forward_wrapper,
          "ConvTranspose2d forward (CUDA)",
          pybind11::arg("input"),
          pybind11::arg("weight"),
          pybind11::arg("bias"),
          pybind11::arg("stride"),
          pybind11::arg("padding"),
          pybind11::arg("dilation"));
}
