#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// Combined CUDA kernel that uses shared memory for weights and tiling/unrolling for input accumulation
__global__ void conv_transpose2d_forward_kernel_combined(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

  // Compute output spatial indices
  int out_w = blockIdx.x * blockDim.x + threadIdx.x;
  int out_h = blockIdx.y * blockDim.y + threadIdx.y;
  int bo_idx = blockIdx.z; // encodes both batch and output channel
  int o = bo_idx % out_channels;
  int b = bo_idx / out_channels;
  
  if (out_w >= out_width || out_h >= out_height)
    return;

  // Allocate shared memory for the weight tile for the corresponding output channel 'o'
  extern __shared__ float shared_weight[]; // size: in_channels * kernel_size * kernel_size

  int weight_count = in_channels * kernel_size * kernel_size;
  int tid = threadIdx.y * blockDim.x + threadIdx.x;
  for (int i = tid; i < weight_count; i += blockDim.x * blockDim.y) {
    int tmp = i;
    int q = tmp % kernel_size;
    tmp /= kernel_size;
    int p = tmp % kernel_size;
    int c = tmp / kernel_size;
    // Map 4D weight index [c, o, p, q] into linear index
    int weight_idx = ((c * out_channels + o) * kernel_size + p) * kernel_size + q;
    shared_weight[i] = weight[weight_idx];
  }
  __syncthreads();

  // Initialize the accumulation with the bias value, using __ldg for read-only caching
  float result = __ldg(&bias[o]);

  // Use tiling for the input channels with a fixed TILE_SIZE to unroll loops and use registers
  const int TILE_SIZE = 4;
  for (int c_base = 0; c_base < in_channels; c_base += TILE_SIZE) {
    float temp_results[TILE_SIZE] = {0.0f, 0.0f, 0.0f, 0.0f};

    #pragma unroll
    for (int p = 0; p < kernel_size; p++) {
      int h_unscaled = out_h + padding - p * dilation;
      if (h_unscaled % stride != 0)
        continue;
      int h_in = h_unscaled / stride;
      if (h_in < 0 || h_in >= in_height)
        continue;

      #pragma unroll
      for (int q = 0; q < kernel_size; q++) {
        int w_unscaled = out_w + padding - q * dilation;
        if (w_unscaled % stride != 0)
          continue;
        int w_in = w_unscaled / stride;
        if (w_in < 0 || w_in >= in_width)
          continue;

        #pragma unroll
        for (int i = 0; i < TILE_SIZE; i++) {
          int c = c_base + i;
          if (c < in_channels) {
            int input_idx = ((b * in_channels + c) * in_height + h_in) * in_width + w_in;
            float input_val = __ldg(&input[input_idx]);
            int weight_tile_idx = c * (kernel_size * kernel_size) + p * kernel_size + q;
            float weight_val = shared_weight[weight_tile_idx];
            temp_results[i] += input_val * weight_val;
          }
        }
      }
    }

    #pragma unroll
    for (int i = 0; i < TILE_SIZE; i++) {
      if (c_base + i < in_channels)
        result += temp_results[i];
    }
  }

  int output_idx = ((b * out_channels + o) * out_height + out_h) * out_width + out_w;
  output[output_idx] = result;
}

// Launcher function
torch::Tensor conv_transpose2d_forward_cuda_combined(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {

  int batch_size = input.size(0);
  int in_channels = input.size(1);
  int in_height = input.size(2);
  int in_width = input.size(3);

  int out_channels = weight.size(1);
  int kernel_size = weight.size(2);  // assume square kernel

  // Calculate output dimensions
  int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  int out_width  = (in_width - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;

  auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());

  // Configure a 2D block for spatial dimensions and use grid.z to cover (batch x out_channels)
  dim3 block(16, 16);
  dim3 grid((out_width + block.x - 1) / block.x,
            (out_height + block.y - 1) / block.y,
            batch_size * out_channels);

  // Shared memory size for weights
  size_t shared_mem_size = in_channels * kernel_size * kernel_size * sizeof(float);

  conv_transpose2d_forward_kernel_combined<<<grid, block, shared_mem_size>>>(
      input.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      output.data_ptr<float>(),
      batch_size,
      in_channels,
      out_channels,
      in_height,
      in_width,
      kernel_size,
      out_height,
      out_width,
      stride,
      padding,
      dilation);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in conv_transpose2d_forward_kernel_combined: %s\n", hipGetErrorString(err));
  }

  return output;
}

// Wrapper function that handles the possibility that bias may be None
torch::Tensor conv_transpose2d_forward_wrapper_combined(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,
    int stride,
    int padding,
    int dilation) {

  int out_channels = weight.size(1);
  torch::Tensor bias;
  if (bias_obj.is(pybind11::none())) {
    bias = torch::zeros({out_channels}, weight.options());
  } else {
    bias = bias_obj.cast<torch::Tensor>();
  }

  return conv_transpose2d_forward_cuda_combined(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &conv_transpose2d_forward_wrapper_combined,
        "ConvTranspose2d forward combined shared-memory and tiling optimization (CUDA)",
        pybind11::arg("input"),
        pybind11::arg("weight"),
        pybind11::arg("bias"),
        pybind11::arg("stride"),
        pybind11::arg("padding"),
        pybind11::arg("dilation"));
}
