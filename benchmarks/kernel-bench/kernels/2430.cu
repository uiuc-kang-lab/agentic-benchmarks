#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Define tile sizes
#define BLOCK_SIZE_M 32  // Output tile height (each block computes 32 rows)
#define BLOCK_SIZE_N 16  // Output tile width (each block computes 16 columns)
#define BLOCK_SIZE_K 16  // Reduction tile depth

// Hybrid Kernel: Combines tiling and warp-level primitives
// A is (K x M): element A[k, m] = A[k * M + m]
// B is (N x K): element B[n, k] = B[n * K + k]
// C is (M x N): element C[m, n] = C[m * N + n]

template <typename scalar_t>
__global__ void matmul_transpose_hybrid_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ C,
    int M,
    int N,
    int K) {

    // Determine the starting indices for this block's tile in C
    int m_start = blockIdx.y * BLOCK_SIZE_M;  // row start in C
    int n_start = blockIdx.x * BLOCK_SIZE_N;  // col start in C

    // Thread indices within the block
    int tx = threadIdx.x; // Expected range: [0, 15]
    int ty = threadIdx.y; // Expected range: [0, 15]

    // Each thread computes two rows: row0 and row1
    int row0 = m_start + tx;             // first row computed by this thread
    int row1 = row0 + (BLOCK_SIZE_M / 2);  // second row computed (offset by 16)
    int col = n_start + ty;              // column index in C

    // Accumulators for the two output elements
    scalar_t acc0 = 0;
    scalar_t acc1 = 0;

    // Declare shared memory tiles
    __shared__ scalar_t A_tile[BLOCK_SIZE_K][BLOCK_SIZE_M]; // Size: 16 x 32
    __shared__ scalar_t B_tile[BLOCK_SIZE_N][BLOCK_SIZE_K];   // Size: 16 x 16

    int numTiles = (K + BLOCK_SIZE_K - 1) / BLOCK_SIZE_K;
    for (int tile = 0; tile < numTiles; tile++) {
        // Load A tile into shared memory
        int kd = threadIdx.x;
        int md = threadIdx.y;
        int global_m = m_start + md;
        int global_k = tile * BLOCK_SIZE_K + kd;
        if (global_m < M && global_k < K)
            A_tile[kd][md] = A[global_k * M + global_m];
        else
            A_tile[kd][md] = 0;

        // Load B tile into shared memory
        int nd = threadIdx.x;
        kd = threadIdx.y;
        int global_n = n_start + nd;
        global_k = tile * BLOCK_SIZE_K + kd;
        if (global_n < N && global_k < K)
            B_tile[nd][kd] = B[global_n * K + global_k];
        else
            B_tile[nd][kd] = 0;

        __syncthreads();

        // Compute the partial results for this tile
        for (int k = 0; k < BLOCK_SIZE_K; k++) {
            scalar_t a_val0 = A_tile[k][tx];                     // for row0
            scalar_t a_val1 = A_tile[k][tx + (BLOCK_SIZE_M / 2)];  // for row1
            scalar_t b_val = B_tile[ty][k];
            acc0 += a_val0 * b_val;
            acc1 += a_val1 * b_val;
        }
        __syncthreads();
    }

    // Perform warp-level reduction using __shfl_down_sync to sum partial results
    for (int offset = 16; offset > 0; offset /= 2) {
        acc0 += __shfl_down_sync(0xffffffff, acc0, offset);
        acc1 += __shfl_down_sync(0xffffffff, acc1, offset);
    }

    // Write the results to global memory
    if (tx == 0) {
        if (row0 < M && col < N) {
            C[row0 * N + col] = acc0;
        }
        if (row1 < M && col < N) {
            C[row1 * N + col] = acc1;
        }
    }
}

// PyTorch binding

torch::Tensor matmul_transpose_cuda(torch::Tensor A, torch::Tensor B) {
    // Dimensions:
    // A: (K x M), B: (N x K), therefore C: (M x N)
    int K = A.size(0);
    int M = A.size(1);
    int N = B.size(0);

    auto C = torch::empty({M, N}, A.options());

    // Define block dimensions: use 16x16 threads per block
    dim3 threads(16, 16);
    // Grid dimensions based on tile sizes
    dim3 blocks((N + BLOCK_SIZE_N - 1) / BLOCK_SIZE_N, (M + BLOCK_SIZE_M - 1) / BLOCK_SIZE_M);

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matmul_transpose_hybrid_kernel", ([&] {
        matmul_transpose_hybrid_kernel<scalar_t><<<blocks, threads>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, N, K);
    }));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_transpose_cuda, "Matrix multiplication with transposed inputs using hybrid kernel (CUDA)");
}
