#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void kl_div_kernel(
    const float* log_predictions,
    const float* targets, 
    float* output,
    const int n) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ float partial_sums[];
    
    float sum = 0.0f;
    
    while (idx < n) {
        float log_pred = log_predictions[idx];
        float target = targets[idx];
        sum += expf(log_pred) - target * log_pred;
        idx += blockDim.x * gridDim.x;
    }
    
    // Warp-level reduction using shuffle intrinsics
    unsigned int mask = 0xffffffff;
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(mask, sum, offset);
    }
    
    // Write reduced value of each warp to shared memory
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        partial_sums[threadIdx.x / warpSize] = sum;
    }
    __syncthreads();

    // Final reduction of each warp's result
    int lane = threadIdx.x;
    int numWarps = blockDim.x / warpSize;
    if (lane < numWarps) {
        sum = partial_sums[lane];
        for (int offset = warpSize/2; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(mask, sum, offset);
        }
        if (lane == 0) {
            atomicAdd(output, sum);
        }
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    const int threads = 128;
    const int blocks = (n + threads - 1) / threads;
    const int shared_mem = threads * sizeof(float);
    
    kl_div_kernel<<<blocks, threads, shared_mem>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward (CUDA)");
}