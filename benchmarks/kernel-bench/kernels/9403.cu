#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

__global__ void optimized_streamed_stride_conv2d_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* output,
    int batch_size,
    int in_channels,
    int input_height,
    int input_width,
    int out_channels,
    int kernel_h,
    int kernel_w,
    int height_out,
    int width_out,
    int stride,
    int pad_h,
    int pad_w,
    int dilation_h,
    int dilation_w) {

    int w_out = blockIdx.x * blockDim.x + threadIdx.x;
    int h_out = blockIdx.y * blockDim.y + threadIdx.y;
    int oc    = blockIdx.z;
    int b     = blockIdx.w;

    if (w_out >= width_out || h_out >= height_out || oc >= out_channels || b >= batch_size) return;

    float sum = (bias != nullptr) ? bias[oc] : 0.0f;

    for (int ic = 0; ic < in_channels; ++ic) {
        for (int kh = 0; kh < kernel_h; ++kh) {
            for (int kw = 0; kw < kernel_w; ++kw) {
                int h_in = h_out * stride + kh * dilation_h - pad_h;
                int w_in = w_out * stride + kw * dilation_w - pad_w;
                if (h_in >= 0 && h_in < input_height && w_in >= 0 && w_in < input_width) {
                    int x_idx = b * in_channels * input_height * input_width + 
                                ic * (input_height * input_width) +
                                h_in * input_width + w_in;
                    int w_idx = oc * in_channels * kernel_h * kernel_w + 
                                ic * kernel_h * kernel_w +
                                kh * kernel_w + kw;
                    sum += x[x_idx] * weight[w_idx];
                }
            }
        }
    }

    int out_idx = b * out_channels * height_out * width_out + 
                  oc * (height_out * width_out) + 
                  h_out * width_out + w_out;
    output[out_idx] = sum;
}

// Forward function using a single kernel launch with batch dimension included in grid

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,  
    int stride,
    std::tuple<int, int> padding,
    std::tuple<int, int> dilation) {

    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "weight must be contiguous");

    const float* bias_ptr = nullptr;
    if (bias.has_value()) {
        TORCH_CHECK(bias->is_cuda(), "bias must be a CUDA tensor");
        TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
        bias_ptr = bias->data_ptr<float>();
    }

    int batch_size   = x.size(0);
    int in_channels  = x.size(1);
    int input_height = x.size(2);
    int input_width  = x.size(3);

    int out_channels = weight.size(0);
    int kernel_h     = weight.size(2);
    int kernel_w     = weight.size(3);

    int pad_h      = std::get<0>(padding);
    int pad_w      = std::get<1>(padding);
    int dilation_h = std::get<0>(dilation);
    int dilation_w = std::get<1>(dilation);

    int height_out = (input_height + 2 * pad_h - dilation_h * (kernel_h - 1) - 1) / stride + 1;
    int width_out  = (input_width  + 2 * pad_w - dilation_w * (kernel_w - 1) - 1) / stride + 1;

    auto output = torch::empty({batch_size, out_channels, height_out, width_out}, x.options());

    dim3 threads(16, 16);
    dim3 blocks(
        (width_out + threads.x - 1) / threads.x,
        (height_out + threads.y - 1) / threads.y,
        out_channels,
        batch_size
    );

    optimized_streamed_stride_conv2d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        input_height,
        input_width,
        out_channels,
        kernel_h,
        kernel_w,
        height_out,
        width_out,
        stride,
        pad_h,
        pad_w,
        dilation_h,
        dilation_w
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Conv2D forward (CUDA)");
}