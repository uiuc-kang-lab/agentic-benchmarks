#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stdexcept>

// Device function for finding minimum value and index
template <typename scalar_t>
__device__ __forceinline__ void find_min(const scalar_t* data, int k, int inner_size, scalar_t& min_val, int& min_index) {
    scalar_t curr_val = data[k * inner_size];
    if (curr_val < min_val) {
        min_val = curr_val;
        min_index = k;
    }
}

template <typename scalar_t>
__global__ void argmin_kernel(const scalar_t* __restrict__ x,
                              int64_t* __restrict__ output,
                              int K,
                              int64_t outer_size,
                              int64_t inner_size) {
    int64_t idx = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    int64_t total_slices = outer_size * inner_size;
    if (idx >= total_slices) return;

    int64_t outer = idx / inner_size;
    int64_t inner = idx % inner_size;
    const scalar_t* slice_start = x + outer * (static_cast<int64_t>(K) * inner_size) + inner;

    scalar_t min_val = slice_start[0];
    int min_index = 0;

    #pragma unroll
    for (int k = 1; k < K; ++k) {
        find_min<scalar_t>(slice_start, k, inner_size, min_val, min_index);
    }
    output[idx] = min_index;
}

at::Tensor argmin_cuda_forward(const at::Tensor &x, int64_t dim) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be a CUDA tensor");

    int dims = x.dim();
    if (dim < 0) {
        dim += dims;
    }
    TORCH_CHECK(dim >= 0 && dim < dims, "Reduction dim out of range");

    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= x.size(i);
    }
    int K = static_cast<int>(x.size(dim));
    int64_t inner_size = 1;
    for (int i = dim + 1; i < dims; i++) {
        inner_size *= x.size(i);
    }

    std::vector<int64_t> out_sizes;
    for (int i = 0; i < dims; i++) {
        if (i == dim)
            continue;
        out_sizes.push_back(x.size(i));
    }
    auto output = at::empty(out_sizes, x.options().dtype(at::kLong));

    int64_t total_slices = outer_size * inner_size;
    int threads = 512;
    int blocks = (total_slices + threads - 1) / threads;

    AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, x.scalar_type(), "argmin_cuda_forward", ([&] {
        const scalar_t* x_data = x.data_ptr<scalar_t>();
        int64_t* output_data = output.data_ptr<int64_t>();
        argmin_kernel<scalar_t><<<blocks, threads>>>(x_data, output_data, K, outer_size, inner_size);
    }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("CUDA kernel failed: ") + hipGetErrorString(err));
    }
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &argmin_cuda_forward, "Argmin forward (CUDA)");
}