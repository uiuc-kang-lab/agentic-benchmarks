#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Tile dimensions for output spatial blocking
constexpr int TILE_DIM = 16;
// Partition size for input channels to enable reduction across blocks
constexpr int CHANNEL_TILE = 16;

// This kernel partitions the convolution reduction over the input channel dimension.
// Each block processes a subset of input channels (a partition) for a particular output element,
// and the partial sums are combined using atomicAdd only when more than one partition is present.
// The bias is added only by the first partition (partition_id == 0).
__global__ void conv2d_atomic_kernel(const float* __restrict__ input,
                                      const float* __restrict__ weight,
                                      const float* __restrict__ bias,
                                      float* __restrict__ output,
                                      int N, int Cin, int H, int W,
                                      int Cout, int K,
                                      int outH, int outW,
                                      int stride, int padding,
                                      int channel_tile, int partitions) {
    // Decode blockIdx.z into a triplet: batch index (n), output channel (cout), and partition id
    int partition_id = blockIdx.z % partitions;
    int n_cout = blockIdx.z / partitions;  // n_cout in [0, N * Cout)
    int n = n_cout / Cout;
    int cout = n_cout % Cout;

    // Determine output spatial coordinates
    int ox = blockIdx.x * TILE_DIM + threadIdx.x;
    int oy = blockIdx.y * TILE_DIM + threadIdx.y;

    if (ox < outW && oy < outH) {
        float partial = 0.0f;
        int cin_start = partition_id * channel_tile;
        int cin_end = (cin_start + channel_tile < Cin) ? (cin_start + channel_tile) : Cin;

        // Accumulate over the partitioned subset of input channels and kernel window
        for (int cin = cin_start; cin < cin_end; ++cin) {
            for (int i = 0; i < K; i++) {
                for (int j = 0; j < K; j++) {
                    int in_y = oy * stride - padding + i;
                    int in_x = ox * stride - padding + j;
                    if (in_y >= 0 && in_y < H && in_x >= 0 && in_x < W) {
                        int input_idx = ((n * Cin + cin) * H + in_y) * W + in_x;
                        int weight_idx = ((cout * Cin + cin) * K + i) * K + j;
                        partial += input[input_idx] * weight[weight_idx];
                    }
                }
            }
        }
        // Only the first partition adds the bias
        if (partition_id == 0 && bias != nullptr) {
            partial += bias[cout];
        }

        int out_idx = ((n * Cout + cout) * outH + oy) * outW + ox;
        // If there are multiple partitions, use atomicAdd to combine partial sums
        if (partitions > 1) {
            atomicAdd(&output[out_idx], partial);
        } else {
            output[out_idx] = partial;
        }
    }
}

// The forward function partitions the reduction over input channels, launching multiple blocks
// per (n, cout) output element if necessary. Atomic operations are used only when more than
// one partition contributes to the same output element, thereby minimizing atomic contention.

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation, // dilation is not used in this implementation
    int groups) {

    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }
    TORCH_CHECK(groups == 1, "groups != 1 not supported by this kernel");

    int N = x.size(0);
    int Cin = x.size(1);
    int H = x.size(2);
    int W = x.size(3);
    int Cout = weight.size(0);
    int K = weight.size(2); // assuming square kernel
    int outH = (H + 2 * padding - K) / stride + 1;
    int outW = (W + 2 * padding - K) / stride + 1;

    // Determine the number of partitions over the input channel dimension
    int partitions = (Cin + CHANNEL_TILE - 1) / CHANNEL_TILE;

    auto output = torch::zeros({N, Cout, outH, outW}, x.options());

    dim3 blockDim(TILE_DIM, TILE_DIM);
    // gridDim.z encodes (n, cout, partition) to allow multiple blocks contributing to the same output element
    dim3 gridDim((outW + TILE_DIM - 1) / TILE_DIM,
                 (outH + TILE_DIM - 1) / TILE_DIM,
                 N * Cout * partitions);

    const float* bias_ptr = (bias.has_value() ? bias.value().data_ptr<float>() : nullptr);

    conv2d_atomic_kernel<<<gridDim, blockDim>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        output.data_ptr<float>(),
        N, Cin, H, W,
        Cout, K,
        outH, outW,
        stride, padding,
        CHANNEL_TILE, partitions);

    hipDeviceSynchronize();

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA forward function for 2D convolution with atomic reduction on input channel partitions");
}
