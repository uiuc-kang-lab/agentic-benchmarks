#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <type_traits>

// Store threshold constants in constant memory
__constant__ float c_upper_threshold_float = 20.0f;
__constant__ float c_lower_threshold_float = -20.0f;
__constant__ double c_upper_threshold_double = 20.0;
__constant__ double c_lower_threshold_double = -20.0;

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_softplus(const scalar_t x) {
    if constexpr (std::is_same<scalar_t, float>::value) {
        if (x > c_upper_threshold_float) return x;
        if (x < c_lower_threshold_float) return expf(x);
        return log1pf(expf(x));
    } else {
        if (x > c_upper_threshold_double) return x;
        if (x < c_lower_threshold_double) return exp(x);
        return log1p(exp(x));
    }
}

template <typename scalar_t>
__global__ void softplus_kernel_optimized(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int size) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Process 4 elements per thread
    #pragma unroll
    for (; idx < size; idx += stride * 4) {
        scalar_t vals[4];
        
        // Coalesced loading of 4 values
        #pragma unroll
        for (int i = 0; i < 4 && idx + i * stride < size; i++) {
            vals[i] = input[idx + i * stride];
        }
        
        // Compute softplus while waiting for memory operations
        #pragma unroll
        for (int i = 0; i < 4 && idx + i * stride < size; i++) {
            output[idx + i * stride] = compute_softplus(vals[i]);
        }
    }
}

torch::Tensor softplus_cuda_forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int size = input.numel();
    const int threads = 256;
    const int blocks = std::min(65535, (size + threads * 4 - 1) / (threads * 4));

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "softplus_forward_cuda", ([&] {
        softplus_kernel_optimized<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &softplus_cuda_forward, "Softplus forward (CUDA)");
}