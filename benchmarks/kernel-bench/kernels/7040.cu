#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <c10/cuda/CUDAStream.h>

template <typename scalar_t>
__device__ __forceinline__ scalar_t reduce_min_along_dim(
    const scalar_t* __restrict__ input,
    const int base,
    const int r,
    const int inner,
    const int outer_idx,
    const int inner_idx) {
  scalar_t min_val = input[base];
  #pragma unroll
  for (int j = 1; j < r; j++) {
    int index = outer_idx * (r * inner) + j * inner + inner_idx;
    scalar_t curr = input[index];
    min_val = curr < min_val ? curr : min_val;
  }
  return min_val;
}

template <typename scalar_t>
__global__ void min_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int outer,
    const int r,
    const int inner) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int total = outer * inner;
  if (idx >= total) return;

  const int outer_idx = idx / inner;
  const int inner_idx = idx % inner;
  const int base = outer_idx * (r * inner) + inner_idx;

  output[idx] = reduce_min_along_dim(
    input, base, r, inner, outer_idx, inner_idx);
}

torch::Tensor forward(torch::Tensor input, int64_t dim) {
  TORCH_CHECK(input.is_cuda(), "input must be a CUDA tensor");
  if (!input.is_contiguous()) {
    input = input.contiguous();
  }

  int ndim = input.dim();
  TORCH_CHECK(dim >= 0 && dim < ndim, "dim out of range");

  int outer = 1;
  for (int i = 0; i < dim; i++) {
    outer *= input.size(i);
  }
  int r = input.size(dim);
  int inner = 1;
  for (int i = dim + 1; i < ndim; i++) {
    inner *= input.size(i);
  }

  std::vector<int64_t> output_shape;
  for (int i = 0; i < ndim; i++) {
    if (i != dim) {
      output_shape.push_back(input.size(i));
    }
  }
  
  auto output = torch::empty(output_shape, input.options());

  const int total = outer * inner;
  const int threads = 256;
  const int blocks = (total + threads - 1) / threads;

  AT_DISPATCH_ALL_TYPES(input.scalar_type(), "min_reduce_cuda", ([&] {
    min_reduce_kernel<scalar_t><<<blocks, threads, 0, 
      c10::cuda::getCurrentCUDAStream().stream()>>>(
        input.data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>(),
        outer,
        r,
        inner);
  }));

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "Min reduction over a specified dimension (CUDA)");
}