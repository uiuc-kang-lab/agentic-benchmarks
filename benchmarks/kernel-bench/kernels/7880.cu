#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

__global__ void conv2d_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    float* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int out_channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int stride,
    const int padding) {

    const int out_h = (height + 2 * padding - kernel_h) / stride + 1;
    const int out_w = (width + 2 * padding - kernel_w) / stride + 1;
    const int total_elements = batch_size * out_channels * out_h * out_w;
    
    // Calculate thread and grid information
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_threads = gridDim.x * blockDim.x;
    
    // Each thread processes multiple elements with stride
    for (int idx = tid; idx < total_elements; idx += total_threads) {
        // Calculate output indices
        const int oc = idx / (batch_size * out_h * out_w);
        const int rem1 = idx % (batch_size * out_h * out_w);
        const int b = rem1 / (out_h * out_w);
        const int rem2 = rem1 % (out_h * out_w);
        const int h = rem2 / out_w;
        const int w = rem2 % out_w;

        float sum = 0.0f;

        // Calculate effective kernel bounds
        const int h_start = max(0, -h * stride + padding);
        const int h_end = min(kernel_h, height - h * stride + padding);
        const int w_start = max(0, -w * stride + padding);
        const int w_end = min(kernel_w, width - w * stride + padding);

        // Compute convolution
        #pragma unroll 4
        for (int ic = 0; ic < in_channels; ++ic) {
            for (int kh = h_start; kh < h_end; ++kh) {
                const int h_in = h * stride + kh - padding;
                #pragma unroll 4
                for (int kw = w_start; kw < w_end; ++kw) {
                    const int w_in = w * stride + kw - padding;
                    const float input_val = __ldg(&input[
                        ((b * in_channels + ic) * height + h_in) * width + w_in]);
                    const float weight_val = __ldg(&weight[
                        ((oc * in_channels + ic) * kernel_h + kh) * kernel_w + kw]);
                    sum += input_val * weight_val;
                }
            }
        }

        output[((b * out_channels + oc) * out_h + h) * out_w + w] = sum;
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {

    TORCH_CHECK(x.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "Weight must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "Input must be contiguous");
    TORCH_CHECK(weight.is_contiguous(), "Weight must be contiguous");

    if (dilation != 1 || groups != 1) {
        return torch::conv2d(x, weight, bias,
                           {stride, stride}, {padding, padding}, {dilation, dilation}, groups);
    }

    auto batch_size = x.size(0);
    auto in_channels = x.size(1);
    auto height = x.size(2);
    auto width = x.size(3);
    auto out_channels = weight.size(0);
    auto kernel_h = weight.size(2);
    auto kernel_w = weight.size(3);

    const int out_h = (height + 2 * padding - kernel_h) / stride + 1;
    const int out_w = (width + 2 * padding - kernel_w) / stride + 1;

    auto output = torch::zeros({batch_size, out_channels, out_h, out_w}, x.options());

    // Optimize thread configuration for H100
    const int threads_per_block = 256;
    const int max_blocks = 512;
    const int num_elements = batch_size * out_channels * out_h * out_w;
    const int num_blocks = min(max_blocks, (num_elements + threads_per_block - 1) / threads_per_block);

    conv2d_kernel<<<num_blocks, threads_per_block>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        out_channels,
        height,
        width,
        kernel_h,
        kernel_w,
        stride,
        padding
    );

    if (bias.has_value()) {
        output.add_(bias.value().view({1, -1, 1, 1}));
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA 2D Convolution");
}