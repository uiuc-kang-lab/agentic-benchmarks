#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void optimized_swish_kernel(const float* __restrict__ x, float* __restrict__ y, int64_t n) {
    const int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int64_t stride = blockDim.x * gridDim.x;

    for (int64_t i = idx; i < n; i += stride) {
        const float val = x[i];
        const float sigmoid = __frcp_rn(1.0f + expf(-val));  // Use reciprocal function to improve performance
        y[i] = val * sigmoid;
    }
}

torch::Tensor optimized_swish_forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    auto y = torch::empty_like(x);
    const int64_t n = x.numel();
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;

    optimized_swish_kernel<<<blocks, threads>>>(x.data_ptr<float>(), y.data_ptr<float>(), n);

    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &optimized_swish_forward, "Optimized Swish activation forward pass (CUDA)");
}