#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel that uses warp-level primitives to perform reduction over the specified dimension.

template <typename scalar_t>
__global__ void warp_sum_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t reduce_size,
    int64_t inner_size,
    int64_t total_output) {

    // Each block handles one output element (one (outer, inner) pair).
    int idx = blockIdx.x;  // index for output element
    if (idx >= total_output) return;

    // Determine corresponding outer and inner indices
    int outer_idx = idx / inner_size;
    int inner_idx = idx % inner_size;

    scalar_t sum = 0;
    // Use lane id of the warp; assume blockDim.x == warpSize (32 threads)
    int lane = threadIdx.x;

    // Each thread in the warp sums elements from the reduction dim in a strided manner
    for (int i = lane; i < reduce_size; i += warpSize) {
        int64_t offset = outer_idx * reduce_size * inner_size + i * inner_size + inner_idx;
        sum += input[offset];
    }

    // Use warp-level shuffle to reduce the partial sums within the warp
    unsigned int mask = 0xffffffff;
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(mask, sum, offset);
    }

    // The first lane writes the result
    if (lane == 0) {
        output[idx] = sum;
    }
}

// Host function wrapping the kernel launch

torch::Tensor sum_reduce_cuda(torch::Tensor input, int64_t dim) {
    // Adjust negative dimensions
    if (dim < 0) dim += input.dim();
    auto sizes = input.sizes().vec();
    int64_t reduce_size = sizes[dim];

    // Compute outer_size: product of dimensions before the reduction dim
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }

    // Compute inner_size: product of dimensions after the reduction dim
    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }

    // Prepare output tensor with reduce dimension set to 1
    sizes[dim] = 1;
    auto output = torch::empty(sizes, input.options());

    // Total number of output elements
    int64_t total_output = outer_size * inner_size;

    // Launch one warp (32 threads) per output element
    const int threads = 32;  // warp size
    const int blocks = total_output;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sum_reduce_cuda", ([&] {
        warp_sum_reduce_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            reduce_size,
            inner_size,
            total_output
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &sum_reduce_cuda, "Sum reduction forward (CUDA) using warp-level primitives");
}
