#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <iostream>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    torch::Tensor C = torch::zeros({M, N}, A.options());

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Create CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipblasSetStream(handle, stream);

    // Perform matrix multiplication using cuBLAS with stream
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B.data_ptr<float>(), N, A.data_ptr<float>(), K, &beta, C.data_ptr<float>(), N);

    // Synchronize the stream
    hipStreamSynchronize(stream);

    // Destroy CUDA stream
    hipStreamDestroy(stream);

    // Destroy cuBLAS handle
    hipblasDestroy(handle);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication with CUDA stream (CUDA)");
}