#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32
#define THREAD_TILE 2
#define UNROLL_FACTOR 8
#define MAX_MATRIX_DIM 8192

// Constant memory for matrix dimensions and number of tiles
__constant__ int d_N;
__constant__ int d_num_tiles;

// Combined kernel using stride loops, __ldg, and unrolling
__global__ void matmul_kernel_strided(const float* __restrict__ A,
                                      const float* __restrict__ B,
                                      float* __restrict__ C) {
    // Shared memory for A and B tiles
    __shared__ float s_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float s_B[BLOCK_SIZE][BLOCK_SIZE];

    // Thread indices
    int ty = threadIdx.y;
    int tx = threadIdx.x;
    // Calculate starting row and column indices for this block's threads
    int blockRow = blockIdx.y * BLOCK_SIZE;
    int blockCol = blockIdx.x * BLOCK_SIZE;

    // Register array to store the output for a 2x2 tile
    float regC[THREAD_TILE][THREAD_TILE] = { {0.0f, 0.0f}, {0.0f, 0.0f} };

    int stride = BLOCK_SIZE / THREAD_TILE;

    // Loop over tiles in the k-dimension with striding
    for (int t = 0; t < d_num_tiles; t++) {
        // Load A tile into shared memory using striding
        for (int i = ty; i < BLOCK_SIZE; i += stride) {
            for (int j = tx; j < BLOCK_SIZE; j += stride) {
                int aRow = blockRow + i;
                int aCol = t * BLOCK_SIZE + j;
                s_A[i][j] = (aRow < d_N && aCol < d_N) ? __ldg(&A[aRow * d_N + aCol]) : 0.0f;
            }
        }

        // Load B tile into shared memory using striding
        for (int i = ty; i < BLOCK_SIZE; i += stride) {
            for (int j = tx; j < BLOCK_SIZE; j += stride) {
                int bRow = t * BLOCK_SIZE + i;
                int bCol = blockCol + j;
                s_B[i][j] = (bRow < d_N && bCol < d_N) ? __ldg(&B[bRow * d_N + bCol]) : 0.0f;
            }
        }

        __syncthreads();

        // Compute the product of the loaded tiles
        for (int k = 0; k < BLOCK_SIZE; k += UNROLL_FACTOR) {
            #pragma unroll
            for (int u = 0; u < UNROLL_FACTOR; u++) {
                float a0 = s_A[ty * THREAD_TILE + 0][k + u];
                float a1 = s_A[ty * THREAD_TILE + 1][k + u];
                float b0 = s_B[k + u][tx * THREAD_TILE + 0];
                float b1 = s_B[k + u][tx * THREAD_TILE + 1];

                regC[0][0] += a0 * b0;
                regC[0][1] += a0 * b1;
                regC[1][0] += a1 * b0;
                regC[1][1] += a1 * b1;
            }
        }

        __syncthreads();
    }

    // Write back the result with boundary checks
    for (int i = 0; i < THREAD_TILE; i++) {
        for (int j = 0; j < THREAD_TILE; j++) {
            int outRow = blockRow + ty * THREAD_TILE + i;
            int outCol = blockCol + tx * THREAD_TILE + j;
            if (outRow < d_N && outCol < d_N) {
                C[outRow * d_N + outCol] = regC[i][j];
            }
        }
    }
}

// C++ interface (Pybind11 binding)
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D matrices");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must have the same dimensions");
    TORCH_CHECK(A.size(0) <= MAX_MATRIX_DIM, "Matrix dimension exceeds maximum supported size");

    int N = A.size(0);
    int num_tiles = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    hipMemcpyToSymbol(HIP_SYMBOL(d_N), &N, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_tiles), &num_tiles, sizeof(int));

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, A.get_device());
    auto C = torch::zeros({N, N}, options);

    dim3 threads(BLOCK_SIZE / THREAD_TILE, BLOCK_SIZE / THREAD_TILE);
    dim3 blocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matmul_kernel_strided<<<blocks, threads>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>());
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Strided Combined Unrolled and Aligned Matrix Multiplication (CUDA)");
}
