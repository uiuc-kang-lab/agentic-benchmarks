#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Constant memory for frequently accessed parameters
__constant__ int c_kernel_size;
__constant__ int c_stride;
__constant__ int c_padding;

__global__ void avg_pool3d_optimized_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int batch_size, int channels,
    int in_d, int in_h, int in_w,
    int out_d, int out_h, int out_w) {
    
    // 2D block configuration with shared memory
    const int tid_x = threadIdx.x;
    const int tid_y = threadIdx.y;
    const int idx_x = blockIdx.x * blockDim.x + tid_x;
    const int idx_y = blockIdx.y * blockDim.y + tid_y;
    
    // Calculate batch and channel indices
    const int batch_channel_idx = blockIdx.z;
    const int n = batch_channel_idx / channels;
    const int c = batch_channel_idx % channels;
    
    // Early exit conditions
    if (idx_x >= out_w || idx_y >= out_h || n >= batch_size)
        return;

    // Pre-calculate kernel volume once
    const float kernel_volume = static_cast<float>(c_kernel_size * c_kernel_size * c_kernel_size);
    const int base_idx = ((n * channels + c) * in_d);
    
    // Process multiple depth slices per thread using grid-stride loop
    for (int d_out = 0; d_out < out_d; d_out++) {
        // Calculate window boundaries
        const int d_start = d_out * c_stride - c_padding;
        const int h_start = idx_y * c_stride - c_padding;
        const int w_start = idx_x * c_stride - c_padding;
        
        // Compute clamped boundaries using branchless max/min
        const int d0 = max(0, d_start);
        const int h0 = max(0, h_start);
        const int w0 = max(0, w_start);
        const int d1 = min(d_start + c_kernel_size, in_d);
        const int h1 = min(h_start + c_kernel_size, in_h);
        const int w1 = min(w_start + c_kernel_size, in_w);
        
        float sum = 0.0f;
        
        #pragma unroll 3
        for (int d = d0; d < d1; ++d) {
            const int d_offset = (base_idx + d) * in_h;
            #pragma unroll 3
            for (int h = h0; h < h1; ++h) {
                const int h_offset = (d_offset + h) * in_w;
                #pragma unroll 3
                for (int w = w0; w < w1; ++w) {
                    sum += __ldg(&input[h_offset + w]);
                }
            }
        }
        
        const int out_idx = (((n * channels + c) * out_d + d_out) * out_h + idx_y) * out_w + idx_x;
        output[out_idx] = sum / kernel_volume;
    }
}

at::Tensor forward(at::Tensor input, int kernel_size, int stride, int padding) {
    TORCH_CHECK(input.dim() == 5, "Input tensor must be 5-dimensional");
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");
    
    const int batch_size = input.size(0);
    const int channels = input.size(1);
    const int in_d = input.size(2);
    const int in_h = input.size(3);
    const int in_w = input.size(4);
    
    const int out_d = (in_d + 2 * padding - kernel_size) / stride + 1;
    const int out_h = (in_h + 2 * padding - kernel_size) / stride + 1;
    const int out_w = (in_w + 2 * padding - kernel_size) / stride + 1;
    
    // Copy constants to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(c_kernel_size), &kernel_size, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(c_stride), &stride, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(c_padding), &padding, sizeof(int));
    
    auto output = at::empty({batch_size, channels, out_d, out_h, out_w}, input.options());
    
    // Optimized block configuration
    dim3 threads(32, 16);  // 512 threads per block
    dim3 blocks(
        (out_w + threads.x - 1) / threads.x,
        (out_h + threads.y - 1) / threads.y,
        batch_size * channels
    );
    
    avg_pool3d_optimized_kernel<<<blocks, threads>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size, channels,
        in_d, in_h, in_w,
        out_d, out_h, out_w
    );
    
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed : ", hipGetErrorString(err));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "3D Average Pooling forward (CUDA)");
}