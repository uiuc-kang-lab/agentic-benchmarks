#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/util/Optional.h>

// Helper device functions for convolution
__device__ inline bool compute_input_position(
    int out_pos, int k, int stride, int dilation, int padding, int L_in, int* in_pos) {
    int pos = out_pos * stride - padding + k * dilation;
    if (pos >= 0 && pos < L_in) {
        *in_pos = pos;
        return true;
    }
    return false;
}

__device__ inline int get_input_index(
    int n, int in_ch, int L_in, int C_in, int in_pos) {
    return n * (C_in * L_in) + in_ch * L_in + in_pos;
}


namespace py = pybind11;

// -----------------------------------------------------
// Optimized 1D convolution CUDA kernel using shared memory with necessary synchronization
// -----------------------------------------------------
__global__ void conv1d_forward_kernel_shared(
    const float* __restrict__ x,
    const float* __restrict__ w,
    const float* __restrict__ bias_ptr, // can be null if no bias
    float* __restrict__ y,
    const int N,         // batch size
    const int C_in,      // input channels
    const int L_in,      // input length
    const int C_out,     // output channels
    const int K,         // kernel size
    const int stride,
    const int padding,
    const int dilation,
    const int groups,
    const int L_out,     // output length
    const int group_size_in,
    const int group_size_out
)
{
    extern __shared__ float shared_w[];

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total = N * C_out * L_out;
    if (idx >= total) return;

    const int out_pos = idx % L_out;
    const int out_ch = (idx / L_out) % C_out;
    const int n = idx / (L_out * C_out);
    
    const int group_idx = out_ch / group_size_out;
    float val = 0.0f;

    // Load weights into shared memory
    if (threadIdx.x < group_size_in * K) {
        shared_w[threadIdx.x] = w[out_ch * (group_size_in * K) + threadIdx.x];
    }
    __syncthreads();

    for (int local_in_ch = 0; local_in_ch < group_size_in; ++local_in_ch) {
        const int in_ch = group_idx * group_size_in + local_in_ch;
        
        for (int k = 0; k < K; ++k) {
            int in_pos;
            if (compute_input_position(out_pos, k, stride, dilation, padding, L_in, &in_pos)) {
                const float x_val = x[get_input_index(n, in_ch, L_in, C_in, in_pos)];
                const float w_val = shared_w[local_in_ch * K + k];
                val += x_val * w_val;
            }
        }
    }

    if (bias_ptr) {
        val += bias_ptr[out_ch];
    }

    y[idx] = val;
}

at::Tensor conv1d_forward_impl_shared(
    const at::Tensor& x,
    const at::Tensor& weight,
    c10::optional<at::Tensor> bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups
)
{
    TORCH_CHECK(x.is_cuda() && weight.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(x.scalar_type() == at::kFloat, "x must be float32");

    const int64_t N = x.size(0);
    const int64_t C_in = x.size(1);
    const int64_t L_in = x.size(2);
    const int64_t C_out = weight.size(0);
    const int64_t K = weight.size(2);
    
    const int64_t L_out = (L_in + 2 * padding - dilation * (K - 1) - 1) / stride + 1;
    TORCH_CHECK(L_out > 0, "Invalid output length");

    auto y = torch::empty({N, C_out, L_out}, x.options());
    const float* bias_ptr = bias_opt.has_value() ? bias_opt->data_ptr<float>() : nullptr;

    const int group_size_in = C_in / groups;
    const int group_size_out = C_out / groups;

    const int total_threads = N * C_out * L_out;
    const int block_size = 256;
    const int grid_size = (total_threads + block_size - 1) / block_size;

    const int shared_mem_size = group_size_in * K * sizeof(float);
    
    conv1d_forward_kernel_shared<<<grid_size, block_size, shared_mem_size>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        y.data_ptr<float>(),
        N, C_in, L_in, C_out, K,
        stride, padding, dilation, groups, L_out,
        group_size_in, group_size_out
    );

    TORCH_CHECK(hipGetLastError() == hipSuccess, "Kernel launch failed");
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", 
        [](at::Tensor x, at::Tensor weight, py::object bias_obj,
           int64_t stride, int64_t padding, int64_t dilation, int64_t groups) {
            c10::optional<at::Tensor> bias;
            if (!bias_obj.is_none()) {
                bias = bias_obj.cast<at::Tensor>();
            }
            return conv1d_forward_impl_shared(x, weight, bias, stride, padding, dilation, groups);
        }, "Optimized 1D Convolution forward (CUDA) with shared memory and necessary synchronization"
    );
}