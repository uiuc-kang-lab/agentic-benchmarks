#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void elu_kernel_optimized(const float* x, float* out, float alpha, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = idx; i < n; i += stride) {
        float val = x[i];
        out[i] = (val > 0) ? val : alpha * (expf(val) - 1);
    }
}

torch::Tensor elu_cuda(torch::Tensor x, float alpha) {
    CHECK_INPUT(x);
    auto out = torch::empty_like(x);
    int n = x.numel();
    
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;
    
    elu_kernel_optimized<<<blocks, threads>>>(x.data_ptr<float>(), out.data_ptr<float>(), alpha, n);
    
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &elu_cuda, "ELU activation with optimized indexing (CUDA)");
}
