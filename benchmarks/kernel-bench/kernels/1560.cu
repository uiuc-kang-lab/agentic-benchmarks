#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// This kernel computes the upper triangular matrix multiplication (C = A * B)
// where only elements with row <= col are evaluated. It uses __ldg() for read-only
// accesses to matrices A and B to exploit the read-only data cache and uses stride
// loops to handle workloads larger than the number of available threads.

__global__ void strided_ldg_upper_triangular_kernel(const float* __restrict__ A,
                                                     const float* __restrict__ B,
                                                     float* __restrict__ C,
                                                     int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N && row <= col) {
        float sum = 0.0f;
        // Loop from k = row to k = col in order to follow the upper triangular property
        for (int k = row; k <= col; k += blockDim.x) {
            if (k < N) {
                // Using __ldg() optimizes global memory loads via the read-only cache.
                float a_val = __ldg(&A[row * N + k]);
                float b_val = __ldg(&B[k * N + col]);
                sum += a_val * b_val;
            }
        }
        C[row * N + col] = sum;
    }
}

// Host function, exposed via pybind11, that wraps the kernel invocation
// It creates a zero tensor for C, launches the kernel, and returns C.

torch::Tensor strided_ldg_upper_triangular_matmul(torch::Tensor A, torch::Tensor B) {
    int N = A.size(0);
    auto C = torch::zeros_like(A);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    strided_ldg_upper_triangular_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &strided_ldg_upper_triangular_matmul, "Strided LDG optimized upper triangular matrix multiplication");
}