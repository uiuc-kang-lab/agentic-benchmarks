#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Optimized Swish kernel ensuring even workload distribution using grid-stride loop
__global__ void swish_kernel_optimized(const float* __restrict__ x, float* __restrict__ y, int64_t n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (; idx < n; idx += stride) {
        float v = x[idx];
        float sig = 1.0f / (1.0f + __expf(-v));
        y[idx] = v * sig;
    }
}

torch::Tensor swish_forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    auto y = torch::empty_like(x);
    int64_t n = x.numel();
    const int threads = 512;
    int blocks = (n + threads - 1) / threads;
    blocks = min(blocks, 144 * 2);  // Limit blocks more conservatively for H100

    swish_kernel_optimized<<<blocks, threads>>>(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        n
    );

    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &swish_forward, "Swish activation forward pass (CUDA) with optimized workload distribution");
}
