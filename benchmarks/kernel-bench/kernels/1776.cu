#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32

// Declare constant memory for tiles of input matrices
__constant__ float A_const[TILE_SIZE * TILE_SIZE];
__constant__ float B_const[TILE_SIZE * TILE_SIZE];

__global__ void triangular_mm_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    const int N,
    const int tile_idx) {
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N) {
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            float sum = 0.f;
            
            // Calculate tile boundaries
            int tile_start = tile_idx * TILE_SIZE;
            int tile_end = min(tile_start + TILE_SIZE, row + 1);
            
            // Use constant memory for the current tile
            for (int k = max(tile_start, col); k < tile_end; ++k) {
                int local_k = k - tile_start;
                sum += A_const[row * TILE_SIZE + local_k] *
                       B_const[local_k * TILE_SIZE + (col % TILE_SIZE)];
            }
            
            if (tile_idx == 0) {
                C[row * N + col] = sum;
            } else {
                C[row * N + col] += sum;
            }
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::zeros_like(A);

    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threads - 1) / threads, (N + threads - 1) / threads);

    // Process matrix in tiles
    int num_tiles = (N + TILE_SIZE - 1) / TILE_SIZE;
    
    for (int tile = 0; tile < num_tiles; ++tile) {
        // Copy tile data to constant memory
        int tile_start = tile * TILE_SIZE;
        int tile_size = min(TILE_SIZE, N - tile_start);
        
        // Prepare tile data
        auto A_tile = A.slice(1, tile_start, tile_start + tile_size);
        auto B_tile = B.slice(0, tile_start, tile_start + tile_size);
        
        hipMemcpyToSymbol(HIP_SYMBOL(A_const), A_tile.data_ptr<float>(),
                           tile_size * N * sizeof(float));
        hipMemcpyToSymbol(HIP_SYMBOL(B_const), B_tile.data_ptr<float>(),
                           tile_size * N * sizeof(float));

        triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
            A.data_ptr<float>(),
            B.data_ptr<float>(),
            C.data_ptr<float>(),
            N,
            tile
        );
    }

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}