#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void hinge_loss_kernel(const float* predictions, const float* targets, float* output, int n) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < n; idx += blockDim.x * gridDim.x) {
        output[idx] = fmaxf(0.0f, 1.0f - predictions[idx] * targets[idx]);
    }
}

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    CHECK_INPUT(predictions);
    CHECK_INPUT(targets);

    int n = predictions.numel();
    torch::Tensor output = torch::empty_like(predictions);

    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    // Create CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Transfer data to CUDA device in stream
    hinge_loss_kernel<<<blocks, threads, 0, stream>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    // Synchronize stream
    hipStreamSynchronize(stream);

    // Clean up stream
    hipStreamDestroy(stream);

    // Compute the mean of the output tensor
    auto mean = torch::mean(output);
    return mean;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Hinge Loss Forward");
}