#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// This kernel computes the upper triangular matrix multiplication (C = A * B)
// using warp-level primitives to optimize small reductions within a warp.
// It distributes the workload more evenly by having each thread compute multiple
// elements, and uses __shfl_down_sync() to perform reductions within a warp.

__global__ void warp_optimized_upper_triangular_kernel(const float* __restrict__ A,
                                                        const float* __restrict__ B,
                                                        float* __restrict__ C,
                                                        int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col_start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int col = col_start; col < N; col += stride) {
        if (row < N && row <= col) {
            float sum = 0.0f;
            // Loop from k = row to k = col in order to follow the upper triangular property
            for (int k = row; k <= col; ++k) {
                float a_val = __ldg(&A[row * N + k]);
                float b_val = __ldg(&B[k * N + col]);
                sum += a_val * b_val;
            }
            // Use warp-level primitive to reduce within the warp
            for (int offset = 16; offset > 0; offset /= 2) {
                sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
            }
            // Write the result from the first thread in the warp
            if (threadIdx.x % 32 == 0) {
                C[row * N + col] = sum;
            }
        }
    }
}

// Host function, exposed via pybind11, that wraps the kernel invocation
// It creates a zero tensor for C, launches the kernel, and returns C.

torch::Tensor warp_optimized_upper_triangular_matmul(torch::Tensor A, torch::Tensor B) {
    int N = A.size(0);
    auto C = torch::zeros_like(A);

    dim3 threadsPerBlock(32, 8); // Use 8 warps per block
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    warp_optimized_upper_triangular_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &warp_optimized_upper_triangular_matmul, "Warp optimized upper triangular matrix multiplication");
}
