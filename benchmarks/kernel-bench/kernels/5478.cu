#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_outputs = batch_size * channels * output_height * output_width;
    if (output_idx >= total_outputs) return;

    // Calculate position using warp-aligned indexing
    const int ow = output_idx % output_width;
    const int oh = (output_idx / output_width) % output_height;
    const int c = (output_idx / (output_width * output_height)) % channels;
    const int b = output_idx / (output_width * output_height * channels);

    // Pre-compute valid ranges for pooling window
    const int ih_start = max(oh * stride - padding, 0);
    const int ih_end = min(oh * stride - padding + kernel_size * dilation, input_height);
    const int iw_start = max(ow * stride - padding, 0);
    const int iw_end = min(ow * stride - padding + kernel_size * dilation, input_width);

    // Base offset for current batch and channel
    const int base_offset = b * (channels * input_height * input_width) +
                           c * (input_height * input_width);

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    // Iterate only over valid positions
    for (int ih = ih_start; ih < ih_end; ih += dilation) {
        const int row_offset = ih * input_width;
        #pragma unroll 4
        for (int iw = iw_start; iw < iw_end; iw += dilation) {
            const int input_idx = base_offset + row_offset + iw;
            max_val = max(max_val, input[input_idx]);
        }
    }

    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    // Use multiple of warp size for thread block size
    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}