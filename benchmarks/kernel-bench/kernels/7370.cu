#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define NUM_STREAMS 4
#define BLOCK_SIZE 256

__global__ void conv2d_cuda_kernel_stream(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int N, int C_in, int H_in, int W_in,
    int C_out, int H_out, int W_out,
    int K_h, int K_w,
    int stride_h, int stride_w,
    int padding_h, int padding_w,
    int dilation_h, int dilation_w,
    int groups,
    int stream_offset,
    int stream_elements
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= stream_elements) return;

    int global_tid = stream_offset + tid;
    
    int w_out = global_tid % W_out;
    int tmp = global_tid / W_out;
    int h_out = tmp % H_out;
    tmp = tmp / H_out;
    int c_out = tmp % C_out;
    int n = tmp / C_out;

    float value = (bias != nullptr) ? bias[c_out] : 0.0f;

    int group = c_out / (C_out / groups);
    int c_in_start = group * (C_in / groups);
    int c_in_end = c_in_start + (C_in / groups);

    #pragma unroll 4
    for (int c_in = c_in_start; c_in < c_in_end; ++c_in) {
        for (int k_h = 0; k_h < K_h; ++k_h) {
            int h_in = h_out * stride_h - padding_h + k_h * dilation_h;
            if (h_in >= 0 && h_in < H_in) {
                for (int k_w = 0; k_w < K_w; ++k_w) {
                    int w_in = w_out * stride_w - padding_w + k_w * dilation_w;
                    if (w_in >= 0 && w_in < W_in) {
                        int input_idx = ((n * C_in + c_in) * H_in + h_in) * W_in + w_in;
                        int weight_idx = (((c_out * (C_in / groups) + (c_in - c_in_start)) * K_h + k_h) * K_w) + k_w;
                        value += input[input_idx] * weight[weight_idx];
                    }
                }
            }
        }
    }

    int output_idx = ((n * C_out + c_out) * H_out + h_out) * W_out + w_out;
    output[output_idx] = value;
}

torch::Tensor conv2d_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias_opt,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> dilation,
    int64_t groups
) {
    input = input.contiguous();
    weight = weight.contiguous();

    TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA");

    const int64_t N = input.size(0);
    const int64_t C_in = input.size(1);
    const int64_t H_in = input.size(2);
    const int64_t W_in = input.size(3);
    const int64_t C_out = weight.size(0);
    const int64_t K_h = weight.size(2);
    const int64_t K_w = weight.size(3);

    const int64_t stride_h = stride[0];
    const int64_t stride_w = stride[1];
    const int64_t padding_h = padding[0];
    const int64_t padding_w = padding[1];
    const int64_t dilation_h = dilation[0];
    const int64_t dilation_w = dilation[1];

    const int64_t H_out = (H_in + 2 * padding_h - dilation_h * (K_h - 1) - 1) / stride_h + 1;
    const int64_t W_out = (W_in + 2 * padding_w - dilation_w * (K_w - 1) - 1) / stride_w + 1;

    auto output = torch::zeros({N, C_out, H_out, W_out}, input.options());

    const float* input_ptr = input.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = nullptr;
    if (bias_opt.has_value()) {
        auto bias = bias_opt.value().contiguous();
        bias_ptr = bias.data_ptr<float>();
    }
    float* output_ptr = output.data_ptr<float>();

    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    const int total_elements = N * C_out * H_out * W_out;
    const int elements_per_stream = (total_elements + NUM_STREAMS - 1) / NUM_STREAMS;
    
    for (int i = 0; i < NUM_STREAMS; i++) {
        const int stream_offset = i * elements_per_stream;
        const int stream_elements = min(elements_per_stream, total_elements - stream_offset);
        
        if (stream_elements <= 0) continue;

        const int threads = BLOCK_SIZE;
        const int blocks = (stream_elements + threads - 1) / threads;

        conv2d_cuda_kernel_stream<<<blocks, threads, 0, streams[i]>>>(
            input_ptr,
            weight_ptr,
            bias_ptr,
            output_ptr,
            N, C_in, H_in, W_in,
            C_out, H_out, W_out,
            K_h, K_w,
            stride_h, stride_w,
            padding_h, padding_w,
            dilation_h, dilation_w,
            groups,
            stream_offset,
            stream_elements
        );
    }

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv2d_cuda, "Streamed 2D convolution (CUDA)",
        py::arg("input"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = std::vector<int64_t>{1, 1},
        py::arg("padding") = std::vector<int64_t>{0, 0},
        py::arg("dilation") = std::vector<int64_t>{1, 1},
        py::arg("groups") = 1
    );
}