#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel that uses warp shuffle intrinsics to perform reduction with uniform control flow
// Each block computes one output element corresponding to a unique (outer, inner) index

template <typename scalar_t>
__global__ void sum_reduce_warp_shuffle_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t reduce_size,
    int64_t inner_size) {

    // Each block is responsible for one output element
    int out_idx = blockIdx.x; // flattened index over outer * inner
    int outer_idx = out_idx / inner_size;
    int inner_idx = out_idx % inner_size;
    int64_t base = outer_idx * reduce_size * inner_size + inner_idx;

    // Each thread accumulates a partial sum over the reduction dimension
    scalar_t sum = 0;
    for (int i = threadIdx.x; i < reduce_size; i += blockDim.x) {
        sum += input[base + i * inner_size];
    }

    // Intra-warp reduction using warp shuffle to avoid divergent branches
    unsigned int mask = 0xffffffff;
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(mask, sum, offset);
    }

    // Use shared memory to accumulate results from different warps
    __shared__ scalar_t warp_sum[32]; // Enough space for up to 32 warps per block
    int warp_id = threadIdx.x / warpSize;
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        warp_sum[warp_id] = sum;
    }
    __syncthreads();

    // First warp now reduces the partial sums from all warps
    int num_warps = (blockDim.x + warpSize - 1) / warpSize;
    if (threadIdx.x < num_warps) {
        sum = warp_sum[threadIdx.x];
        for (int offset = warpSize/2; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(mask, sum, offset);
        }
        if (threadIdx.x == 0) {
            output[out_idx] = sum;
        }
    }
}

// Host function that prepares tensor dimensions and launches the warp-shuffle reduction kernel

torch::Tensor sum_reduce_warp_shuffle_cuda(torch::Tensor input, int64_t dim) {
    // Handle negative dimensions
    if (dim < 0) dim += input.dim();

    // Compute sizes: outer, reduce, and inner
    auto sizes = input.sizes().vec();
    int64_t reduce_size = sizes[dim];
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }
    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }

    // Prepare output tensor shape (with dimension 'dim' collapsed to 1)
    sizes[dim] = 1;
    auto output = torch::empty(sizes, input.options());

    // Launch kernel: one block per output element
    int64_t num_output = outer_size * inner_size;
    int threads = 256;
    int blocks = num_output;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sum_reduce_warp_shuffle_cuda", ([&] {
        sum_reduce_warp_shuffle_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            reduce_size,
            inner_size
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &sum_reduce_warp_shuffle_cuda, "Sum reduction with warp shuffle to minimize divergence (CUDA)");
}
