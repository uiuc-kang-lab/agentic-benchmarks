#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Define tile sizes
#define BLOCK_SIZE_M 32  // Output tile height (each block computes 32 rows)
#define BLOCK_SIZE_N 16  // Output tile width (each block computes 16 columns)
#define BLOCK_SIZE_K 16  // Reduction tile depth

// Kernel: each thread computes a 2x1 sub-tile (2 rows, 1 column) of C
// A is (K x M): element A[k, m] = A[k * M + m]
// B is (N x K): element B[n, k] = B[n * K + k]
// C is (M x N): element C[m, n] = C[m * N + n]

template <typename scalar_t>
__global__ void optimized_thread_block_indexing_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ C,
    int M,
    int N,
    int K) {

    // Calculate position using optimized indexing
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M && col < N) {
        scalar_t sum = 0;
        for (int k = 0; k < K; k++) {
            // A is transposed: (k,M) -> access A[k * M + row]
            // B is transposed: (N,K) -> access B[col * K + k]
            sum += __ldg(&A[k * M + row]) * __ldg(&B[col * K + k]);
        }
        C[row * N + col] = sum;
    }
}

// PyTorch binding
torch::Tensor matmul_transpose_cuda(torch::Tensor A, torch::Tensor B) {
    // Dimensions:
    // A: (K x M), B: (N x K), therefore C: (M x N)
    int K = A.size(0);
    int M = A.size(1);
    int N = B.size(0);

    auto C = torch::empty({M, N}, A.options());

    // Define block dimensions: use 16x16 threads per block
    dim3 threads(16, 16);
    // Grid dimensions based on tile sizes
    dim3 blocks((M + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "optimized_thread_block_indexing_kernel", ([&] {
        optimized_thread_block_indexing_kernel<scalar_t><<<blocks, threads>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, N, K);
    }));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_transpose_cuda, "Optimized matrix multiplication with transposed inputs using efficient thread and block indexing (CUDA)");
}