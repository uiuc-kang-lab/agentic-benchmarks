#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel using balanced workload distribution with 3D grid: gridDim.z covers batch_size*out_channels,
// gridDim.x and gridDim.y cover spatial dimensions output_w and output_h respectively.

__global__ void depthwise_balanced_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int input_h,
    int input_w,
    int out_channels,
    int output_h,
    int output_w,
    int kernel_size,
    int stride,
    int padding,
    int channels_per_group
) {
    // Determine the (batch, channel) index from gridDim.z
    int oc_batch = blockIdx.z;              // combined index for (batch, channel)
    int b = oc_batch / out_channels;
    int oc = oc_batch % out_channels;

    // Determine spatial coordinates for the output
    int w_out = blockIdx.x * blockDim.x + threadIdx.x;
    int h_out = blockIdx.y * blockDim.y + threadIdx.y;

    if (w_out >= output_w || h_out >= output_h) return;

    // Map oc to input channel and weight index
    int in_ch = oc / channels_per_group;
    int weight_ch = oc % channels_per_group;

    float sum = 0.0f;
    
    // Loop over the kernel window
    for (int kh = 0; kh < kernel_size; ++kh) {
        int h_in = h_out * stride + kh - padding;
        for (int kw = 0; kw < kernel_size; ++kw) {
            int w_in = w_out * stride + kw - padding;
            if (h_in >= 0 && h_in < input_h && w_in >= 0 && w_in < input_w) {
                int input_idx = b * (in_channels * input_h * input_w) +
                                in_ch * (input_h * input_w) +
                                h_in * input_w + w_in;
                int weight_idx = in_ch * (channels_per_group * kernel_size * kernel_size) +
                                 weight_ch * (kernel_size * kernel_size) +
                                 kh * kernel_size + kw;
                sum += input[input_idx] * weight[weight_idx];
            }
        }
    }

    if (bias != nullptr) {
        sum += bias[oc];
    }

    int out_idx = b * (out_channels * output_h * output_w) +
                  oc * (output_h * output_w) +
                  h_out * output_w + w_out;
    output[out_idx] = sum;
}


torch::Tensor forward(
    torch::Tensor input,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding
) {
    TORCH_CHECK(input.is_cuda() && weight.is_cuda(), "Inputs must be CUDA tensors");
    if (bias.has_value()) {
        TORCH_CHECK(bias->is_cuda(), "Bias must be a CUDA tensor");
    }
    TORCH_CHECK(input.is_contiguous() && weight.is_contiguous(), "Input and weight must be contiguous");
    if (bias.has_value()) {
        TORCH_CHECK(bias->is_contiguous(), "Bias must be contiguous");
    }
    TORCH_CHECK(weight.dim() == 4, "Weight must be a 4D tensor");

    int batch_size = input.size(0);
    int in_channels = input.size(1);
    int input_h = input.size(2);
    int input_w = input.size(3);
    int kernel_size = weight.size(2);
    int channels_per_group = weight.size(1);
    int out_channels = in_channels * channels_per_group;

    int output_h = (input_h + 2 * padding - kernel_size) / stride + 1;
    int output_w = (input_w + 2 * padding - kernel_size) / stride + 1;

    auto output = torch::empty({batch_size, out_channels, output_h, output_w}, input.options());

    // Set up block and grid dimensions.
    // Use a 2D block for spatial dimensions (e.g. 16x16 threads)
    const int TILE_W = 16;
    const int TILE_H = 16;
    dim3 blockDim(TILE_W, TILE_H, 1);
    
    // Grid dimensions: x covers output_w, y covers output_h, and z covers batch * out_channels
    dim3 gridDim((output_w + TILE_W - 1) / TILE_W,
                 (output_h + TILE_H - 1) / TILE_H,
                 batch_size * out_channels);

    depthwise_balanced_kernel<<<gridDim, blockDim>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.has_value() ? bias->data_ptr<float>() : nullptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        input_h,
        input_w,
        out_channels,
        output_h,
        output_w,
        kernel_size,
        stride,
        padding,
        channels_per_group
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Depthwise 2D Convolution with balanced workload distribution",
          py::arg("input"), py::arg("weight"), py::arg("bias") = py::none(), py::arg("stride"), py::arg("padding"));
}
