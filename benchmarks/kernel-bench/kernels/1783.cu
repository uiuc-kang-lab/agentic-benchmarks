#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel to compute C = tril(A * B) for lower triangular matrices A and B using shared memory.
__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                       const float* __restrict__ B,
                                       float* __restrict__ C,
                                       int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Allocate shared memory for a tile of A and B
    __shared__ float As[16][16];
    __shared__ float Bs[16][16];

    float sum = 0.f;

    // Loop over tiles of A and B
    for (int t = 0; t < (N + 15) / 16; ++t) {
        // Load data into shared memory
        if (row < N && t * 16 + threadIdx.x < N)
            As[threadIdx.y][threadIdx.x] = A[row * N + t * 16 + threadIdx.x];
        else
            As[threadIdx.y][threadIdx.x] = 0.f;

        if (col < N && t * 16 + threadIdx.y < N)
            Bs[threadIdx.y][threadIdx.x] = B[(t * 16 + threadIdx.y) * N + col];
        else
            Bs[threadIdx.y][threadIdx.x] = 0.f;

        __syncthreads();

        // Compute partial product
        for (int k = 0; k < 16; ++k) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write result to C
    if (row < N && col < N) {
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            C[row * N + col] = sum;
        }
    }
}

// C++ interface exposed to PyTorch.
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    const int threads = 16;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threads - 1) / threads, (N + threads - 1) / threads);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication with shared memory (CUDA)");
}