#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void matmul_thread_efficiency_kernel(const float* __restrict__ A,
                                                 const float* __restrict__ B,
                                                 float* __restrict__ C,
                                                 int M, int N, int K) {
    // Calculate the row and column index each warp/block will compute
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float c_value = 0.0f;
        // Each thread computes one element in the result matrix
        for (int k = 0; k < K; k += 4) {
            float4 a = *(float4*)&A[row * K + k];
            float4 b = *(float4*)&B[col * K + k];
            c_value += a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w;
            c_value += A[row * K + k] * B[col * K + k];
        }
        C[row * N + col] = c_value;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.dim() == 2, "A must be 2D");
    TORCH_CHECK(B.dim() == 2, "B must be 2D");
    TORCH_CHECK(A.size(1) == B.size(1), "A and B must have the same K dimension");
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be on CUDA");
    TORCH_CHECK(A.is_contiguous() && B.is_contiguous(), "Inputs must be contiguous");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(0);

    auto C = torch::empty({M, N}, A.options());

    // Optimize the grid dimensions for better memory access patterns
    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);

    matmul_thread_efficiency_kernel<<<grid, block>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, N, K
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Thread/bock optimized matrix multiplication with transposed B (CUDA)");
}