#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32

// Optimized tiled matrix multiplication kernel that minimizes __syncthreads() calls.
// It only synchronizes when necessary: after loading into shared memory and before starting a new tile iteration.
__global__ void matmul_min_sync_kernel(const float* A, const float* B, float* C, int M, int K, int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    float sum = 0.0f;
    int numTiles = (K + TILE_SIZE - 1) / TILE_SIZE;

    for (int t = 0; t < numTiles; t++) {
        // Load tile from A into shared memory
        int a_col = t * TILE_SIZE + tx;
        if (row < M && a_col < K) {
            As[ty][tx] = A[row * K + a_col];
        } else {
            As[ty][tx] = 0.0f;
        }

        // Load tile from B into shared memory
        int b_row = t * TILE_SIZE + ty;
        if (col < N && b_row < K) {
            Bs[ty][tx] = B[b_row * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }

        // Ensure the tile is loaded before computing
        if (t < numTiles - 1) __syncthreads();

        // Compute the dot product for the current tile
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += As[ty][k] * Bs[k][tx];
        }

        // Synchronize only if there is another tile to process
        if (t < numTiles - 1) {
            if (t < numTiles - 1) __syncthreads();
        }
    }

    // Write the computed value to global memory
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

// Host function called from PyTorch
torch::Tensor matmul_cuda(torch::Tensor A, torch::Tensor B) {
    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    auto C = torch::zeros({M, N}, A.options());

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);

    matmul_min_sync_kernel<<<grid, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        M, K, N
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_cuda, "Matrix multiplication with minimized synchronizations (CUDA)");
}
