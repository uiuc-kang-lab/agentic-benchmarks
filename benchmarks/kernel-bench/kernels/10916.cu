#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static const int BLOCK_SIZE = 256;
static const int WARP_SIZE = 32;

template <typename scalar_t>
__global__ void mse_forward_kernel(
    const scalar_t* __restrict__ preds,
    const scalar_t* __restrict__ tgts,
    double* __restrict__ sum_out,
    const int64_t num_elements
) {
    __shared__ double shm[BLOCK_SIZE];
    const unsigned int tid = threadIdx.x;
    const unsigned int wid = tid / WARP_SIZE;
    const unsigned int lane = tid % WARP_SIZE;
    
    // Align global memory access to warp size
    unsigned int idx = (blockIdx.x * BLOCK_SIZE + tid);
    const unsigned int grid_stride = gridDim.x * BLOCK_SIZE;
    double thread_sum = 0.0;

    // Process elements in warp-aligned chunks
    #pragma unroll 4
    for (; idx + WARP_SIZE <= num_elements; idx += grid_stride) {
        double pred = static_cast<double>(preds[idx]);
        double tgt = static_cast<double>(tgts[idx]);
        double diff = pred - tgt;
        thread_sum += diff * diff;
    }
    
    // Handle remaining elements (always executed by full warp)
    if (idx < num_elements) {
        double pred = static_cast<double>(preds[idx]);
        double tgt = static_cast<double>(tgts[idx]);
        double diff = pred - tgt;
        thread_sum += diff * diff;
    }

    // Warp-level reduction first
    #pragma unroll
    for (int offset = WARP_SIZE/2; offset > 0; offset >>= 1) {
        thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
    }

    // First thread in each warp writes to shared memory
    if (lane == 0) {
        shm[wid] = thread_sum;
    }
    __syncthreads();

    // Final reduction across warps
    if (tid < (BLOCK_SIZE / WARP_SIZE)) {
        double warp_sum = shm[tid];
        if (tid == 0) {
            for (int i = 1; i < (BLOCK_SIZE / WARP_SIZE); ++i) {
                warp_sum += shm[i];
            }
            atomicAdd(sum_out, warp_sum);
        }
    }
}

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.is_cuda(), "predictions must be a CUDA tensor");
    TORCH_CHECK(targets.is_cuda(), "targets must be a CUDA tensor");
    TORCH_CHECK(predictions.numel() == targets.numel(),
                "predictions and targets must have the same number of elements");

    const int64_t num_elements = predictions.numel();
    auto accumulator = torch::zeros({1}, predictions.options().dtype(at::kDouble));

    // Ensure grid size is multiple of warps for better alignment
    const int grid_size = ((num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE + WARP_SIZE - 1) & ~(WARP_SIZE - 1);

    AT_DISPATCH_FLOATING_TYPES(predictions.scalar_type(), "mse_forward_cuda", [&] {
        mse_forward_kernel<scalar_t><<<grid_size, BLOCK_SIZE>>>(
            predictions.data_ptr<scalar_t>(),
            targets.data_ptr<scalar_t>(),
            accumulator.data_ptr<double>(),
            num_elements
        );
    });

    auto result = accumulator.div_(static_cast<double>(num_elements));
    return result.to(predictions.dtype());
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Mean Squared Error (MSE) forward (CUDA)");
}