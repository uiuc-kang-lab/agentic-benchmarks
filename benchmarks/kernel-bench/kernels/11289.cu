#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Warp-level reduction using shuffle intrinsics
__inline__ __device__ float warpReduceSum(float val) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__global__ void hybrid_cosine_similarity_loss_kernel(
    const float* __restrict__ predictions,
    const float* __restrict__ targets,
    float* output,
    int N,
    int D) {
    
    // Use shared memory for larger reductions
    extern __shared__ float s_data[];
    float* s_dot = s_data;
    float* s_pred_sq = s_data + blockDim.x;
    float* s_target_sq = s_pred_sq + blockDim.x;
    
    int row = blockIdx.x;
    int tid = threadIdx.x;
    int stride = blockDim.x;
    int lane = tid & (warpSize - 1);
    int warp_id = tid >> 5;

    const float* pred_row = predictions + row * D;
    const float* target_row = targets + row * D;

    float sum_dot = 0.0f;
    float sum_pred_sq = 0.0f;
    float sum_target_sq = 0.0f;

    // Coalesced memory access with stride
    for (int i = tid; i < D; i += stride) {
        float p = pred_row[i];
        float t = target_row[i];
        sum_dot += p * t;
        sum_pred_sq += p * p;
        sum_target_sq += t * t;
    }

    // First do warp-level reduction
    sum_dot = warpReduceSum(sum_dot);
    sum_pred_sq = warpReduceSum(sum_pred_sq);
    sum_target_sq = warpReduceSum(sum_target_sq);

    // Store warp results to shared memory
    if (lane == 0) {
        s_dot[warp_id] = sum_dot;
        s_pred_sq[warp_id] = sum_pred_sq;
        s_target_sq[warp_id] = sum_target_sq;
    }
    __syncthreads();

    // Final reduction using shared memory only for the first warp
    if (tid < warpSize) {
        int num_warps = (blockDim.x + warpSize - 1) / warpSize;
        float final_dot = (tid < num_warps) ? s_dot[tid] : 0.0f;
        float final_pred_sq = (tid < num_warps) ? s_pred_sq[tid] : 0.0f;
        float final_target_sq = (tid < num_warps) ? s_target_sq[tid] : 0.0f;

        final_dot = warpReduceSum(final_dot);
        final_pred_sq = warpReduceSum(final_pred_sq);
        final_target_sq = warpReduceSum(final_target_sq);

        if (tid == 0) {
            const float eps = 1e-8f;
            float norm_pred = sqrtf(final_pred_sq);
            float norm_target = sqrtf(final_target_sq);
            float denominator = norm_pred * norm_target;
            denominator = fmaxf(denominator, eps);
            float cos_sim = final_dot / denominator;
            atomicAdd(output, (1.0f - cos_sim) / N);
        }
    }
}

torch::Tensor hybrid_cosine_similarity_loss_forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.dim() == 2, "predictions must be 2D");
    TORCH_CHECK(targets.dim() == 2, "targets must be 2D");
    TORCH_CHECK(predictions.sizes() == targets.sizes(), "Input tensors must have the same shape");
    TORCH_CHECK(predictions.scalar_type() == torch::kFloat32, "predictions must be float32");
    TORCH_CHECK(targets.scalar_type() == torch::kFloat32, "targets must be float32");

    int N = predictions.size(0);
    int D = predictions.size(1);

    auto output = torch::zeros({1}, predictions.options());
    const int block_size = 256;
    size_t shared_mem = 3 * block_size * sizeof(float);

    hybrid_cosine_similarity_loss_kernel<<<N, block_size, shared_mem>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        N,
        D
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &hybrid_cosine_similarity_loss_forward, "Hybrid Cosine Similarity Loss Forward (CUDA)");
}