#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>

// Combined kernel using both __ldg() for faster memory access and shared memory for reduced global memory load

template <typename scalar_t>
__global__ void max_pool2d_kernel_combined(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    extern __shared__ scalar_t shared_data[];
    const int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    const int ow = output_idx % output_width;
    const int oh = (output_idx / output_width) % output_height;
    const int c = (output_idx / (output_width * output_height)) % channels;
    const int b = output_idx / (output_width * output_height * channels);

    // Shared memory allocation
    const int shared_mem_offset = threadIdx.x * kernel_size * kernel_size;
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    // Pre-calculate base input index for better memory access pattern
    const int input_batch_offset = b * (channels * input_height * input_width);
    const int input_channel_offset = c * (input_height * input_width);

    #pragma unroll
    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = oh * stride - padding + kh * dilation;
        if (ih >= 0 && ih < input_height) {
            const int ih_offset = ih * input_width;

            #pragma unroll
            for (int kw = 0; kw < kernel_size; kw++) {
                const int iw = ow * stride - padding + kw * dilation;
                if (iw >= 0 && iw < input_width) {
                    const int input_idx = input_batch_offset + input_channel_offset + ih_offset + iw;
                    shared_data[shared_mem_offset + kh * kernel_size + kw] = __ldg(&input[input_idx]);
                }
            }
        }
    }
    __syncthreads();

    // Compute max value from shared memory
    #pragma unroll
    for (int i = 0; i < kernel_size * kernel_size; i++) {
        max_val = max(max_val, shared_data[shared_mem_offset + i]);
    }

    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward_combined(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward_combined", ([&] {
        const int shared_memory_size = threads * kernel_size * kernel_size * sizeof(scalar_t);
        max_pool2d_kernel_combined<scalar_t><<<blocks, threads, shared_memory_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward_combined, "Max Pool 2D forward combined optimization (CUDA)");
}