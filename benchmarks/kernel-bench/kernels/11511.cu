#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void kl_div_kernel(
    const float* __restrict__ log_predictions,
    const float* __restrict__ targets, 
    float* __restrict__ output,
    const int n) {
    
    extern __shared__ float partial_sums[];
    float sum = 0.0f;
    
    // Process 4 elements per thread with grid-stride loop
    const int tid = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    for(int i = 0; i < 4; ++i) {
        int idx = tid + i * blockDim.x;
        if(idx < n) {
            float log_pred = log_predictions[idx];
            float target = targets[idx];
            sum += target * (log_pred - logf(target));
        }
    }
    
    // Shared memory reduction
    partial_sums[threadIdx.x] = sum;
    __syncthreads();

    for(int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if(threadIdx.x < stride) {
            partial_sums[threadIdx.x] += partial_sums[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Final warp-level reduction with shuffle
    if(threadIdx.x < 32) {
        float val = partial_sums[threadIdx.x];
        for(int offset = 16; offset > 0; offset >>= 1)
            val += __shfl_down_sync(0xffffffff, val, offset);
        
        if(threadIdx.x == 0)
            atomicAdd(output, val);
    }
}

torch::Tensor kl_div_cuda_forward(
    torch::Tensor log_predictions,
    torch::Tensor targets) {
    
    const int n = log_predictions.numel();
    auto output = torch::zeros({1}, log_predictions.options());
    
    const int threads = 256;
    int sm_count;
hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, 0);
    const int blocks_per_sm = 8;
    int blocks = min((n + threads*4 - 1) / (threads*4), sm_count * blocks_per_sm);
    
    kl_div_kernel<<<blocks, threads, threads * sizeof(float)>>>(
        log_predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );
    
    return output / static_cast<float>(n);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &kl_div_cuda_forward, "KL divergence forward optimized (CUDA)");
}