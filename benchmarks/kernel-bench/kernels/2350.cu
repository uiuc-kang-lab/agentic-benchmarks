#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

const int TILE_SIZE = 16;

__global__ void matmul_transposed_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;

    float c_val = 0.0;

    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        int k_offset = t * TILE_SIZE;

        if (row < M && (k_offset + tx) < K) {
            As[ty][tx] = A[row * K + k_offset + tx];
        } else {
            As[ty][tx] = 0.0;
        }

        if (col < N && (k_offset + ty) < K) {
            Bs[ty][tx] = B[col * K + k_offset + ty];
        } else {
            Bs[ty][tx] = 0.0;
        }

        __syncthreads();

        #pragma unroll
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            c_val = __fmaf_rn(As[ty][k], Bs[k][tx], c_val);
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        C[row * N + col] = c_val;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.dim() == 2, "A must be 2D");
    TORCH_CHECK(B.dim() == 2, "B must be 2D");
    TORCH_CHECK(A.size(1) == B.size(1), "A and B must have same K dimension");
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be on CUDA");
    TORCH_CHECK(A.is_contiguous() && B.is_contiguous(), "Inputs must be contiguous");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(0);

    auto C = torch::empty({M, N}, A.options());
    
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
    dim3 block(TILE_SIZE, TILE_SIZE);
    
    matmul_transposed_kernel<<<grid, block>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, N, K
    );
    
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Kernel failed: ", hipGetErrorString(err));
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Matrix multiplication with transposed B (CUDA)");
}