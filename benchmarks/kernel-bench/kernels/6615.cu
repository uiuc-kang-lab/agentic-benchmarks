#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Declare constant memory for dimension sizes
__constant__ int64_t c_dim_size;

// This kernel assumes the input tensor is conceptually [outer, dim, inner] where:
//  outer_size = product of dimensions before the reduced dimension
//  dim_size   = size of the reduced dimension
//  inner_size = product of dimensions after the reduced dimension
// Each block in the x-dimension handles one outer index, and blocks in the y-dimension tile the inner dimension.

template <typename scalar_t>
__global__ void stride_loop_max_reduce_kernel_with_constant_memory(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t inner_size
) {
    // Determine which outer index this block is working on
    int outer_idx = blockIdx.x;
    
    // Determine the tile index in the inner dimension
    int inner_idx = blockIdx.y * blockDim.x + threadIdx.x;
    if (inner_idx >= inner_size) return;

    // Base offset for this outer index
    int64_t base_offset = outer_idx * c_dim_size * inner_size;

    // Initialize the maximum value with the first element in the reduction dimension
    scalar_t max_val = input[base_offset + inner_idx];

    // Loop over the reduction dimension; note that for each i, the memory access
    // is to a contiguous block of memory for threads in the same warp, ensuring coalescing.
    for (int i = 1; i < c_dim_size; i++) {
        scalar_t val = input[base_offset + i * inner_size + inner_idx];
        max_val = max(max_val, val);
    }

    // Write the result to output. The output tensor is conceptually [outer, inner].
    output[outer_idx * inner_size + inner_idx] = max_val;
}

// This function computes the outer_size and inner_size from the input shape, similar to the reference implementation,
// but then it launches a 2D grid that ensures coalesced memory accesses along the inner dimension.

torch::Tensor stride_loop_max_reduce_cuda_forward_with_constant_memory(torch::Tensor input, int64_t dim) {
    // Handle negative dimension
    if (dim < 0) dim += input.dim();

    // Calculate outer_size: product of sizes before the 'dim' dimension
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= input.size(i);
    }
    
    // Calculate inner_size: product of sizes after the 'dim' dimension
    int64_t inner_size = 1;
    for (int i = dim + 1; i < input.dim(); i++) {
        inner_size *= input.size(i);
    }
    
    // Size along the reduction dimension
    const int64_t dim_size = input.size(dim);

    // Copy dim_size to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(c_dim_size), &dim_size, sizeof(int64_t));

    // Create the output tensor by removing the reduced dimension
    auto output_sizes = input.sizes().vec();
    output_sizes.erase(output_sizes.begin() + dim);
    auto output = torch::empty(output_sizes, input.options());

    // Configure block and grid sizes.
    // Use a 2D grid: grid.x = outer_size; grid.y covers the inner dimension tiled by the block.
    const int threads = 256; // Aligned to 8 warps for better efficiency
    int blocks_y = (inner_size + threads - 1) / threads;
    dim3 grid(outer_size, blocks_y);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "stride_loop_max_reduce_forward_with_constant_memory", ([&] {
        stride_loop_max_reduce_kernel_with_constant_memory<scalar_t><<<grid, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            inner_size
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &stride_loop_max_reduce_cuda_forward_with_constant_memory, "Stride loop Max reduction forward with constant memory (CUDA)");
}