#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__device__ __forceinline__ void calculate_indices(
    const int output_idx,
    const int output_width,
    const int output_height,
    const int channels,
    int& ow,
    int& oh,
    int& c,
    int& b
) {
    ow = output_idx % output_width;
    oh = (output_idx / output_width) % output_height;
    c = (output_idx / (output_width * output_height)) % channels;
    b = output_idx / (output_width * output_height * channels);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_max_pool(
    const scalar_t* input,
    const int b,
    const int c,
    const int oh,
    const int ow,
    const int input_height,
    const int input_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    const int ih_start = oh * stride - padding;
    const int iw_start = ow * stride - padding;
    
    #pragma unroll
    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = ih_start + kh * dilation;
        if (ih >= 0 && ih < input_height) {
            #pragma unroll
            for (int kw = 0; kw < kernel_size; kw++) {
                const int iw = iw_start + kw * dilation;
                if (iw >= 0 && iw < input_width) {
                    const int input_idx = ((b * input_height * channels + c) * input_height + ih) * input_width + iw;
                    max_val = max(max_val, input[input_idx]);
                }
            }
        }
    }
    return max_val;
}

template <typename scalar_t>
__global__ void max_pool2d_kernel(
    const scalar_t* input,
    scalar_t* output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    int ow, oh, c, b;
    calculate_indices<scalar_t>(
        output_idx, output_width, output_height, channels,
        ow, oh, c, b
    );

    output[output_idx] = compute_max_pool<scalar_t>(
        input, b, c, oh, ow,
        input_height, input_width,
        kernel_size, stride, padding, dilation
    );
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}