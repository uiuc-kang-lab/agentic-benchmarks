#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256
#define WARP_SIZE 32

__device__ float get_element(const float* __restrict__ matrix, int row, int col, int ld, bool transpose) {
    if (transpose)
        return matrix[col * ld + row];
    else
        return matrix[row * ld + col];
}

__global__ void matmul_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int M, int N, int K,
    int lda, int ldb, int ldc,
    bool transA, bool transB) {
    
    // Calculate global thread index
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate row and column from linear index
    const int row = tid / N;
    const int col = tid % N;
    
    if (row < M && col < N) {
        float sum = 0.0f;
        
        #pragma unroll 4
        for (int k = 0; k < K; k++) {
            float a_val = get_element(A, row, k, lda, transA);
            float b_val = get_element(B, k, col, ldb, transB);
            sum += a_val * b_val;
        }
        
        C[row * ldc + col] = sum;
    }
}

torch::Tensor matmul_cuda(torch::Tensor A, torch::Tensor B) {
    if (!A.is_cuda() || !B.is_cuda()) {
        throw std::invalid_argument("Input tensors must be on CUDA devices");
    }

    if (A.dim() != 2 || B.dim() != 2) {
        throw std::invalid_argument("Input tensors must be 2D matrices");
    }

    int64_t A_rows = A.size(0);
    int64_t A_cols = A.size(1);
    int64_t B_rows = B.size(0);
    int64_t B_cols = B.size(1);

    bool transA = false;
    bool transB = false;
    int64_t M, N, K;
    int lda, ldb, ldc;

    if (A_rows >= A_cols && B_rows == A_cols) {
        M = A_rows;
        K = A_cols;
        N = B_cols;
        lda = A.stride(0);
        ldb = B.stride(0);
    } else if (A_cols > A_rows && B_rows == A_rows) {
        transA = true;
        M = A_cols;
        K = A_rows;
        N = B_cols;
        lda = A.stride(1);
        ldb = B.stride(0);
    } else if (A_rows >= A_cols && B_cols == A_cols) {
        transB = true;
        M = A_rows;
        K = A_cols;
        N = B_rows;
        lda = A.stride(0);
        ldb = B.stride(1);
    } else if (A_cols > A_rows && B_cols == A_rows) {
        transA = true;
        transB = true;
        M = A_cols;
        K = A_rows;
        N = B_rows;
        lda = A.stride(1);
        ldb = B.stride(1);
    } else {
        throw std::invalid_argument("Incompatible matrix dimensions for multiplication");
    }

    ldc = N;

    auto C = torch::empty({M, N}, A.options());

    // Calculate total number of elements and required blocks
    const int total_elements = M * N;
    const int num_blocks = (total_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Launch kernel with 1D grid and 1D blocks
    matmul_kernel<<<num_blocks, BLOCK_SIZE>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        M, N, K,
        lda, ldb, ldc,
        transA, transB);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_cuda, "Matrix multiplication with tall-and-skinny optimization (CUDA)");
}