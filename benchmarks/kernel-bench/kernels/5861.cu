#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>
#include <cmath>
#include <algorithm>

template <typename scalar_t>
__global__ void tuned_maxpool3d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t* __restrict__ indices,
    const int batch_size,
    const int channels,
    const int input_d, const int input_h, const int input_w,
    const int output_d, const int output_h, const int output_w,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation) {

    // Using optimized block dimensions (16x8)
    const int w_out = blockIdx.x * blockDim.x + threadIdx.x;
    const int h_out = blockIdx.y * blockDim.y + threadIdx.y;
    const int linear_idx = blockIdx.z;

    const int d_out = linear_idx % output_d;
    const int tmp = linear_idx / output_d;
    const int c = tmp % channels;
    const int b = tmp / channels;

    if (w_out >= output_w || h_out >= output_h) return;

    // Compute input window bounds
    const int d_start = d_out * stride - padding;
    const int h_start = h_out * stride - padding;
    const int w_start = w_out * stride - padding;

    // Pre-compute valid ranges for pooling window
    const int k_d_start = max(0, (-d_start + dilation - 1) / dilation);
    const int k_d_end = min(kernel_size, (input_d - d_start + dilation - 1) / dilation);
    const int k_h_start = max(0, (-h_start + dilation - 1) / dilation);
    const int k_h_end = min(kernel_size, (input_h - h_start + dilation - 1) / dilation);
    const int k_w_start = max(0, (-w_start + dilation - 1) / dilation);
    const int k_w_end = min(kernel_size, (input_w - w_start + dilation - 1) / dilation);

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    int max_index = -1;

    // Base index for current batch and channel
    const int batch_channel_offset = (b * channels + c) * input_d;

    #pragma unroll 4
    for (int kd = k_d_start; kd < k_d_end; kd++) {
        const int d_in = d_start + kd * dilation;
        const int d_offset = batch_channel_offset + d_in;

        #pragma unroll 4
        for (int kh = k_h_start; kh < k_h_end; kh++) {
            const int h_in = h_start + kh * dilation;
            const int h_offset = d_offset * input_h + h_in;

            #pragma unroll 4
            for (int kw = k_w_start; kw < k_w_end; kw++) {
                const int w_in = w_start + kw * dilation;
                const int input_idx = (h_offset * input_w) + w_in;
                const scalar_t val = __ldg(&input[input_idx]);
                if (val > max_val) {
                    max_val = val;
                    max_index = input_idx;
                }
            }
        }
    }

    // Compute output index
    const int output_idx = (((b * channels + c) * output_d + d_out) * output_h + h_out) * output_w + w_out;
    output[output_idx] = max_val;
    if (indices != nullptr) {
        indices[output_idx] = max_index;
    }
}

torch::Tensor max_pool3d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation,
    bool return_indices,
    bool ceil_mode) {

    auto input_sizes = input.sizes();
    const int batch_size = input_sizes[0];
    const int channels = input_sizes[1];
    const int input_d = input_sizes[2];
    const int input_h = input_sizes[3];
    const int input_w = input_sizes[4];

    const int output_d = ceil_mode ? 
        ceil((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    const int output_h = ceil_mode ?
        ceil((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    const int output_w = ceil_mode ?
        ceil((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);

    auto output = torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options());
    auto indices = return_indices ? 
        torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options().dtype(torch::kLong)) :
        torch::Tensor();

    // Optimized block dimensions: 16x8 threads (128 threads per block)
    dim3 threads(16, 8);
    dim3 blocks(
        (output_w + threads.x - 1) / threads.x,
        (output_h + threads.y - 1) / threads.y,
        batch_size * channels * output_d
    );

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool3d_forward_cuda", ([&] {
        tuned_maxpool3d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            return_indices ? indices.data_ptr<int64_t>() : nullptr,
            batch_size, channels,
            input_d, input_h, input_w,
            output_d, output_h, output_w,
            kernel_size, stride, padding, dilation);
    }));

    if (return_indices) {
        return torch::stack({output, indices}, 0);
    }
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool3d_cuda_forward, "Tuned Max Pool 3D forward (CUDA)");
}