#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void max_pool2d_kernel_2d_block(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int bz = blockIdx.z;

    const int TILE_WIDTH = 16;
    const int TILE_HEIGHT = 16;

    const int out_x = bx * TILE_WIDTH + tx;
    const int out_y = by * TILE_HEIGHT + ty;
    
    const int batch_idx = bz / channels;
    const int channel_idx = bz % channels;

    if (out_x >= output_width || out_y >= output_height || batch_idx >= batch_size) return;

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    #pragma unroll
    for (int kh = 0; kh < kernel_size; kh++) {
        const int ih = out_y * stride - padding + kh * dilation;
        
        if (ih >= 0 && ih < input_height) {
            #pragma unroll
            for (int kw = 0; kw < kernel_size; kw++) {
                const int iw = out_x * stride - padding + kw * dilation;
                
                if (iw >= 0 && iw < input_width) {
                    const int input_idx = batch_idx * (channels * input_height * input_width) +
                                        channel_idx * (input_height * input_width) +
                                        ih * input_width +
                                        iw;
                    max_val = max(max_val, __ldg(&input[input_idx]));
                }
            }
        }
    }

    const int output_idx = batch_idx * (channels * output_height * output_width) +
                          channel_idx * (output_height * output_width) +
                          out_y * output_width +
                          out_x;
    
    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const dim3 threads(16, 16);
    const dim3 blocks(
        (output_width + threads.x - 1) / threads.x,
        (output_height + threads.y - 1) / threads.y,
        batch_size * channels
    );

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_kernel_2d_block<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}