#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>


namespace py = pybind11;

// Utility function to parse int or sequence of ints from a pybind11 object
inline std::vector<int64_t> parseIntArrayRef(const py::object &obj) {
    std::vector<int64_t> result;
    if (py::isinstance<py::int_>(obj)) {
        result.push_back(obj.cast<int64_t>());
    } else if (py::isinstance<py::sequence>(obj)) {
        for (auto item : obj.cast<py::sequence>()) {
            result.push_back(py::cast<int64_t>(item));
        }
    } else {
        throw std::runtime_error("Expected int or sequence of ints");
    }
    return result;
}

// This kernel assumes that the input, weight, and bias tensors are 128-bit (16-byte) aligned.
// It uses __ldg() for read-only global memory accesses to leverage the read-only data cache,
// which can reduce memory latency on NVIDIA H100 GPUs.

__global__ void conv_transpose2d_kernel_ldg_aligned(
    const float * __restrict__ input,
    const float * __restrict__ weight,
    const float * __restrict__ bias,
    float * __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int h_in,
    const int w_in,
    const int out_channels,
    const int h_out,
    const int w_out,
    const int kernel_size,
    const int stride_h,
    const int stride_w,
    const int padding_h,
    const int padding_w,
    const int groups,
    const int in_channels_per_group,
    const int out_channels_per_group
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch_size * out_channels * h_out * w_out;
    if (index >= total) return;

    // Compute output spatial indices
    int w = index % w_out;
    int tmp = index / w_out;
    int h = tmp % h_out;
    tmp = tmp / h_out;
    int c = tmp % out_channels;
    int n = tmp / out_channels;

    int g = c / out_channels_per_group;
    int c_local = c % out_channels_per_group;

    float sum = 0.0f;

    // Optional: Ensure pointers are assumed to be 16-byte aligned.
    // This hint can help the compiler generate efficient aligned load instructions.
    const float *input_aligned  = input;   // assumed 128-bit aligned
    const float *weight_aligned = weight;  // assumed 128-bit aligned
    const float *bias_aligned   = bias;      // assumed 128-bit aligned

    // Loop over the kernel spatial dimensions with unrolling
    #pragma unroll
    for (int kh = 0; kh < kernel_size; ++kh) {
        #pragma unroll
        for (int kw = 0; kw < kernel_size; ++kw) {
            int h_in_candidate = h + padding_h - kh;
            int w_in_candidate = w + padding_w - kw;
            if ((h_in_candidate % stride_h == 0) && (w_in_candidate % stride_w == 0)) {
                int h_in_idx = h_in_candidate / stride_h;
                int w_in_idx = w_in_candidate / stride_w;
                if (h_in_idx >= 0 && h_in_idx < h_in && w_in_idx >= 0 && w_in_idx < w_in) {
                    // Sum over the input channels in the corresponding group
                    for (int r = 0; r < in_channels_per_group; ++r) {
                        int in_channel = g * in_channels_per_group + r;
                        int input_idx = ((n * in_channels + in_channel) * h_in + h_in_idx) * w_in + w_in_idx;
                        int weight_idx = (((g * in_channels_per_group + r) * out_channels_per_group + c_local) * kernel_size + kh) * kernel_size + kw;
                        // Use __ldg() to load from global memory, assuming 128-bit alignment
                        float in_val = __ldg(&input_aligned[input_idx]);
                        float w_val  = __ldg(&weight_aligned[weight_idx]);
                        sum += in_val * w_val;
                    }
                }
            }
        }
    }

    if (bias_aligned != nullptr) {
        sum += __ldg(&bias_aligned[c]);
    }

    int output_idx = ((n * out_channels + c) * h_out + h) * w_out + w;
    output[output_idx] = sum;
}

// The forward function callable from PyTorch
// It sets up the convolution parameters and launches the CUDA kernel.

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    py::object stride = py::int_(1),
    py::object padding = py::int_(0),
    py::object output_padding = py::int_(0),
    int64_t groups = 1
) {
    auto stride_vec = parseIntArrayRef(stride);
    auto padding_vec = parseIntArrayRef(padding);
    auto output_padding_vec = parseIntArrayRef(output_padding);

    int stride_h = stride_vec[0];
    int stride_w = (stride_vec.size() > 1) ? stride_vec[1] : stride_h;
    int padding_h = padding_vec[0];
    int padding_w = (padding_vec.size() > 1) ? padding_vec[1] : padding_h;
    int output_padding_h = output_padding_vec[0];
    int output_padding_w = (output_padding_vec.size() > 1) ? output_padding_vec[1] : output_padding_h;

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int h_in = x.size(2);
    const int w_in = x.size(3);

    const int kernel_size = weight.size(2); // square kernel assumed
    int out_channels = weight.size(1) * groups;

    // Compute output dimensions for transposed convolution
    int h_out = (h_in - 1) * stride_h - 2 * padding_h + kernel_size + output_padding_h;
    int w_out = (w_in - 1) * stride_w - 2 * padding_w + kernel_size + output_padding_w;

    auto output_tensor = torch::zeros({batch_size, out_channels, h_out, w_out}, x.options());

    int in_channels_per_group = in_channels / groups;
    int out_channels_per_group = out_channels / groups;

    int total_elements = batch_size * out_channels * h_out * w_out;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;

    const float* input_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float* output_ptr = output_tensor.data_ptr<float>();

    conv_transpose2d_kernel_ldg_aligned<<<grid_size, block_size>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch_size,
        in_channels,
        h_in,
        w_in,
        out_channels,
        h_out,
        w_out,
        kernel_size,
        stride_h,
        stride_w,
        padding_h,
        padding_w,
        groups,
        in_channels_per_group,
        out_channels_per_group
    );

    hipDeviceSynchronize();
    return output_tensor;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ConvTranspose2d forward with __ldg and 128-bit aligned accesses",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride") = 1,
          py::arg("padding") = 0,
          py::arg("output_padding") = 0,
          py::arg("groups") = 1);
}
