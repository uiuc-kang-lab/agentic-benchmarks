#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16
#define MATRIX_SIZE_THRESHOLD 512  // Threshold to switch between custom kernel and cuBLAS

// Custom kernel for smaller matrices
__global__ void MatmulKernel(const float* __restrict__ A, const float* __restrict__ B, 
                            float* __restrict__ C, int M, int K, int N) {
    __shared__ float As[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];
    
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    float cValue = 0.0f;

    int numTiles = (K + TILE_WIDTH - 1) / TILE_WIDTH;
    
    for (int t = 0; t < numTiles; t++) {
        int tiledCol = t * TILE_WIDTH + threadIdx.x;
        int tiledRow = t * TILE_WIDTH + threadIdx.y;
        
        As[threadIdx.y][threadIdx.x] = (row < M && tiledCol < K) ? 
            A[row * K + tiledCol] : 0.0f;
        Bs[threadIdx.y][threadIdx.x] = (tiledRow < K && col < N) ? 
            B[tiledRow * N + col] : 0.0f;
        
        __syncthreads();
        
        #pragma unroll
        for (int i = 0; i < TILE_WIDTH; i++) {
            cValue += As[threadIdx.y][i] * Bs[i][threadIdx.x];
        }
        
        __syncthreads();
    }
    
    if (row < M && col < N) {
        C[row * N + col] = cValue;
    }
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.is_contiguous(), "A must be contiguous");
    TORCH_CHECK(B.is_contiguous(), "B must be contiguous");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    auto C = torch::zeros({M, N}, A.options());

    // Choose between custom kernel and cuBLAS based on matrix size
    if (M <= MATRIX_SIZE_THRESHOLD && N <= MATRIX_SIZE_THRESHOLD) {
        // Use custom kernel for smaller matrices
        dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
        dim3 gridDim((N + TILE_WIDTH - 1) / TILE_WIDTH, 
                     (M + TILE_WIDTH - 1) / TILE_WIDTH);
        
        MatmulKernel<<<gridDim, blockDim>>>(
            A.data_ptr<float>(), B.data_ptr<float>(), 
            C.data_ptr<float>(), M, K, N);
    } else {
        // Use cuBLAS for larger matrices
        static hipblasHandle_t handle = nullptr;
        if (handle == nullptr) {
            hipblasCreate(&handle);
        }
        
        float alpha = 1.0f;
        float beta = 0.0f;
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                   N, M, K, &alpha, 
                   B.data_ptr<float>(), N, 
                   A.data_ptr<float>(), K, 
                   &beta, C.data_ptr<float>(), N);
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Adaptive matrix multiplication (CUDA)");
}