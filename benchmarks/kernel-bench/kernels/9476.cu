#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// CUDA kernel using shared memory and exploring optimal block size configurations
__global__ void conv_transpose2d_forward_kernel_optimal_blocks(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {
  // Using blockDim.x * blockDim.y to determine thread position
  int out_w = blockIdx.x * blockDim.x + threadIdx.x;
  int out_h = blockIdx.y * blockDim.y + threadIdx.y;

  int bo_idx = blockIdx.z;
  int o = bo_idx % out_channels;
  int b = bo_idx / out_channels;

  extern __shared__ float shared_weight[]; // size: in_channels * kernel_size * kernel_size

  // Cooperative loading of weights into shared memory
  int weight_count = in_channels * kernel_size * kernel_size;
  int tid = threadIdx.y * blockDim.x + threadIdx.x;
  for (int i = tid; i < weight_count; i += blockDim.x * blockDim.y) {
    int tmp = i;
    int q = tmp % kernel_size;
    tmp /= kernel_size;
    int p = tmp % kernel_size;
    int c = tmp / kernel_size;
    int weight_idx = ((c * out_channels + o) * kernel_size + p) * kernel_size + q;
    shared_weight[i] = weight[weight_idx];
  }
  __syncthreads();

  if (out_w >= out_width || out_h >= out_height)
    return;

  float out_val = bias[o];

  for (int c = 0; c < in_channels; ++c) {
    for (int p = 0; p < kernel_size; ++p) {
      int h_unscaled = out_h + padding - p * dilation;
      if (h_unscaled % stride != 0)
        continue;
      int h_in = h_unscaled / stride;
      if (h_in < 0 || h_in >= in_height)
        continue;
      for (int q = 0; q < kernel_size; ++q) {
        int w_unscaled = out_w + padding - q * dilation;
        if (w_unscaled % stride != 0)
          continue;
        int w_in = w_unscaled / stride;
        if (w_in < 0 || w_in >= in_width)
          continue;
        int input_idx = ((b * in_channels + c) * in_height + h_in) * in_width + w_in;
        int weight_shared_idx = c * (kernel_size * kernel_size) + p * kernel_size + q;
        out_val += input[input_idx] * shared_weight[weight_shared_idx];
      }
    }
  }

  int output_idx = ((b * out_channels + o) * out_height + out_h) * out_width + out_w;
  output[output_idx] = out_val;
}

// CUDA launcher function with different block sizes
torch::Tensor conv_transpose2d_forward_cuda_optimal_blocks(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {

  int batch_size = input.size(0);
  int in_channels = input.size(1);
  int in_height = input.size(2);
  int in_width = input.size(3);

  int out_channels = weight.size(1);
  int kernel_size = weight.size(2);

  int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  int out_width  = (in_width - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;

  auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());

  // Trial block configurations for exploration
  // Configure threads using a variety of potential block dimensions
  // Typical choices can be 256 or 512 which are divisible by the warp size of 32
  const int block_sizes[] = {32, 64, 128, 256, 512};
  int best_block_size = 256; // default, but can test other block sizes to find optimal
  float best_time = FLT_MAX;

  for (int block_size : block_sizes) {
    dim3 block(block_size, 8);
    dim3 grid((out_width + block.x - 1) / block.x,
              (out_height + block.y - 1) / block.y,
              batch_size * out_channels);

    int weight_count = in_channels * kernel_size * kernel_size;
    size_t shared_mem_size = weight_count * sizeof(float);

    // Launch kernel to profile execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    conv_transpose2d_forward_kernel_optimal_blocks<<<grid, block, shared_mem_size>>>(
        input.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        out_channels,
        in_height,
        in_width,
        kernel_size,
        out_height,
        out_width,
        stride,
        padding,
        dilation);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    if (milliseconds < best_time) {
      best_time = milliseconds;
      best_block_size = block_size;
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  // Launch using the best block size determined
  dim3 block(best_block_size, 8);
  dim3 grid((out_width + block.x - 1) / block.x,
            (out_height + block.y - 1) / block.y,
            batch_size * out_channels);

  int weight_count = in_channels * kernel_size * kernel_size;
  size_t shared_mem_size = weight_count * sizeof(float);

  conv_transpose2d_forward_kernel_optimal_blocks<<<grid, block, shared_mem_size>>>(
      input.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      output.data_ptr<float>(),
      batch_size,
      in_channels,
      out_channels,
      in_height,
      in_width,
      kernel_size,
      out_height,
      out_width,
      stride,
      padding,
      dilation);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in conv_transpose2d_forward_kernel_optimal_blocks: %s\n", hipGetErrorString(err));
  }

  return output;
}

// Wrapper function
torch::Tensor conv_transpose2d_forward_wrapper_optimal_blocks(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,
    int stride,
    int padding,
    int dilation) {

  int out_channels = weight.size(1);
  torch::Tensor bias;
  if (bias_obj.is(pybind11::none())) {
    bias = torch::zeros({out_channels}, weight.options());
  } else {
    bias = bias_obj.cast<torch::Tensor>();
  }

  return conv_transpose2d_forward_cuda_optimal_blocks(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &conv_transpose2d_forward_wrapper_optimal_blocks,
        "ConvTranspose2d forward with optimized block size (CUDA)",
        pybind11::arg("input"),
        pybind11::arg("weight"),
        pybind11::arg("bias"),
        pybind11::arg("stride"),
        pybind11::arg("padding"),
        pybind11::arg("dilation"));
}