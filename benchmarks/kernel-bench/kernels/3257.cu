#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>

template <typename scalar_t>
__device__ __forceinline__ scalar_t warp_reduce_max(scalar_t val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = max(val, __shfl_down_sync(0xffffffff, val, offset));
    }
    return val;
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t warp_reduce_sum(scalar_t val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

template <typename scalar_t>
__global__ void log_softmax_forward_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int dim_size) {

    const int batch_idx = blockIdx.x;
    const int tid = threadIdx.x;
    const int lane_id = tid % 32;
    const int warp_id = tid / 32;
    
    extern __shared__ __align__(sizeof(scalar_t)) unsigned char smem[];
    scalar_t* shared_data = reinterpret_cast<scalar_t*>(smem);

    // First pass: find max value
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    #pragma unroll 4
    for (int idx = tid; idx < dim_size; idx += blockDim.x) {
        max_val = max(max_val, __ldg(input + batch_idx * dim_size + idx));
    }

    max_val = warp_reduce_max(max_val);
    if (lane_id == 0) shared_data[warp_id] = max_val;
    __syncthreads();

    if (tid < 32) {
        max_val = (tid < (blockDim.x / 32)) ? shared_data[tid] : -std::numeric_limits<scalar_t>::infinity();
        max_val = warp_reduce_max(max_val);
        if (tid == 0) shared_data[0] = max_val;
    }
    __syncthreads();
    
    max_val = shared_data[0];

    // Second pass: compute sum of exp(x - max_val)
    scalar_t sum = 0;
    #pragma unroll 4
    for (int idx = tid; idx < dim_size; idx += blockDim.x) {
        sum += exp(__ldg(input + batch_idx * dim_size + idx) - max_val);
    }

    sum = warp_reduce_sum(sum);
    if (lane_id == 0) shared_data[warp_id] = sum;
    __syncthreads();

    if (tid < 32) {
        sum = (tid < (blockDim.x / 32)) ? shared_data[tid] : 0;
        sum = warp_reduce_sum(sum);
        if (tid == 0) shared_data[0] = sum;
    }
    __syncthreads();
    
    scalar_t log_sum = log(shared_data[0]);

    // Final pass: compute output
    #pragma unroll 4
    for (int idx = tid; idx < dim_size; idx += blockDim.x) {
        output[batch_idx * dim_size + idx] = 
            (__ldg(input + batch_idx * dim_size + idx) - max_val) - log_sum;
    }
}

torch::Tensor log_softmax_cuda_forward(torch::Tensor input, int64_t dim) {
    auto ndim = input.dim();
    dim = dim >= 0 ? dim : dim + ndim;

    std::vector<int64_t> permute_dims;
    for (int64_t i = 0; i < ndim; ++i) {
        if (i != dim) permute_dims.push_back(i);
    }
    permute_dims.push_back(dim);

    input = input.permute(permute_dims).contiguous();
    auto output = torch::empty_like(input);
    
    int64_t batch_size = input.numel() / input.size(-1);
    int64_t dim_size = input.size(-1);

    const int threads = 256;
    
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "log_softmax_forward_cuda", ([&] {
        size_t shared_mem_size = (threads/32) * sizeof(scalar_t);
        log_softmax_forward_kernel<scalar_t><<<batch_size, threads, shared_mem_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            dim_size);
    }));

    std::vector<int64_t> inverse_permute_dims(ndim);
    for (size_t i = 0; i < permute_dims.size(); ++i) {
        inverse_permute_dims[permute_dims[i]] = i;
    }
    return output.permute(inverse_permute_dims);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &log_softmax_cuda_forward, "LogSoftmax forward (CUDA)");
}