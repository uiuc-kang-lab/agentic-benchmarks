#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define TILE_SIZE 256
#define WARP_SIZE 32

// Constant memory for frequently accessed data
__constant__ float const_data[TILE_SIZE];

__global__ void constant_mem_prod_reduce_kernel(const float* __restrict__ input,
                                                float* __restrict__ output,
                                                const int dim_size,
                                                const int stride) {
    __shared__ float shared_data[TILE_SIZE];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int lane_id = tid & (WARP_SIZE - 1);
    const int warp_id = tid / WARP_SIZE;
    
    // Initialize partial product
    float thread_prod = 1.0f;
    
    // Process input in tiles to maximize shared memory usage
    for (int tile_start = 0; tile_start < dim_size; tile_start += TILE_SIZE) {
        // Reset shared memory for new tile
        shared_data[tid] = 1.0f;
        __syncthreads();
        
        // Load tile into shared memory with coalesced access
        const int tile_end = min(tile_start + TILE_SIZE, dim_size);
        for (int i = tile_start + tid; i < tile_end; i += blockDim.x) {
            shared_data[tid] *= input[bid + i * stride];
        }
        __syncthreads();
        
        // Reduce within tile
        if (tid < WARP_SIZE) {
            float warp_prod = 1.0f;
            // Each thread in first warp reduces a portion of shared memory
            for (int i = tid; i < TILE_SIZE; i += WARP_SIZE) {
                warp_prod *= shared_data[i];
            }
            
            // Warp-level reduction using shuffle
            #pragma unroll
            for (int offset = WARP_SIZE/2; offset > 0; offset >>= 1) {
                warp_prod *= __shfl_down_sync(0xffffffff, warp_prod, offset);
            }
            
            // First thread in warp updates the thread product
            if (lane_id == 0) {
                thread_prod *= warp_prod;
            }
        }
    }
    
    // Final warp-level reduction
    if (tid < WARP_SIZE) {
        float final_prod = __shfl_sync(0xffffffff, thread_prod, 0);
        if (tid == 0) {
            output[bid] = final_prod;
        }
    }
}

torch::Tensor forward(torch::Tensor x, int dim) {
    CHECK_INPUT(x);
    
    auto sizes = x.sizes().vec();
    int dim_size = sizes[dim];
    sizes.erase(sizes.begin() + dim);
    torch::Tensor output = torch::empty(sizes, x.options());
    
    int num_elements = output.numel();
    int stride = x.stride(dim);
    
    const float* input_ptr = x.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();
    
    // Copy frequently accessed data to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(const_data), input_ptr, TILE_SIZE * sizeof(float));

    // Launch configuration optimized for shared memory usage
    int threads = TILE_SIZE;  // Match shared memory tile size
    int blocks = num_elements;
    
    constant_mem_prod_reduce_kernel<<<blocks, threads>>>(input_ptr, output_ptr, dim_size, stride);
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Constant memory optimized product reduction (CUDA)");
}