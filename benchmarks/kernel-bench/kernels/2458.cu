#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void matmul_transpose_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ C,
    const int M,
    const int N,
    const int K) {
    
    const int TILE_SIZE = 32;
    __shared__ scalar_t As[TILE_SIZE][TILE_SIZE];
    __shared__ scalar_t Bs[TILE_SIZE][TILE_SIZE];
    
    const int row = blockIdx.x * TILE_SIZE + threadIdx.x;
    const int col = blockIdx.y * TILE_SIZE + threadIdx.y;
    
    scalar_t sum = 0;
    
    for (int tile = 0; tile < (K + TILE_SIZE - 1) / TILE_SIZE; ++tile) {
        // Load tiles from A (transposed) and B (transposed) into shared memory
        if (row < M && (tile * TILE_SIZE + threadIdx.y) < K)
            As[threadIdx.x][threadIdx.y] = A[(tile * TILE_SIZE + threadIdx.y) * M + row];
        else
            As[threadIdx.x][threadIdx.y] = 0;
            
        if (col < N && (tile * TILE_SIZE + threadIdx.x) < K)
            Bs[threadIdx.y][threadIdx.x] = B[col * K + tile * TILE_SIZE + threadIdx.x];
        else
            Bs[threadIdx.y][threadIdx.x] = 0;
            
        __syncthreads();
        
        // Compute partial dot product for this tile
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[threadIdx.x][k] * Bs[threadIdx.y][k];
        }
        
        __syncthreads();
    }
    
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

torch::Tensor matmul_transpose_cuda(torch::Tensor A, torch::Tensor B) {
    const int K = A.size(0);
    const int M = A.size(1);
    const int N = B.size(0);
    
    auto C = torch::empty({M, N}, A.options());
    
    const int TILE_SIZE = 32;
    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((M + TILE_SIZE - 1) / TILE_SIZE,
                (N + TILE_SIZE - 1) / TILE_SIZE);
    
    AT_DISPATCH_FLOATING_TYPES(A.type(), "matmul_transpose_kernel", ([&] {
        matmul_transpose_kernel<scalar_t><<<blocks, threads>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, N, K
        );
    }));
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_transpose_cuda, "Matrix multiplication with transpose forward (CUDA)");
}