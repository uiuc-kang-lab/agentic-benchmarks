#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

__global__ void conv2d_cuda_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int N, int C_in, int H_in, int W_in,
    int C_out, int H_out, int W_out,
    int K_h, int K_w,
    int stride_h, int stride_w,
    int padding_h, int padding_w,
    int dilation_h, int dilation_w,
    int groups
) {
    extern __shared__ float shared_mem[];
    float* shared_input = shared_mem;
    float* shared_weight = shared_mem + blockDim.x * blockDim.y;

    int w_out = blockIdx.x * blockDim.x + threadIdx.x;
    int h_out = blockIdx.y * blockDim.y + threadIdx.y;
    int c_out = blockIdx.z % C_out;
    int n = blockIdx.z / C_out;

    if (w_out >= W_out || h_out >= H_out || n >= N) return;

    float value = (bias != nullptr) ? bias[c_out] : 0.0f;

    int group = c_out / (C_out / groups);
    int c_in_start = group * (C_in / groups);
    int c_in_end = c_in_start + (C_in / groups);

    for (int c_in = c_in_start; c_in < c_in_end; ++c_in) {
        int h_in = h_out * stride_h - padding_h;
        int w_in = w_out * stride_w - padding_w;

        if (h_in >= 0 && h_in < H_in && w_in >= 0 && w_in < W_in) {
            int input_idx = ((n * C_in + c_in) * H_in + h_in) * W_in + w_in;
            shared_input[threadIdx.y * blockDim.x + threadIdx.x] = input[input_idx];
        }
        __syncthreads();

        for (int k_h = 0; k_h < K_h; ++k_h) {
            for (int k_w = 0; k_w < K_w; ++k_w) {
                int h_in = h_out * stride_h - padding_h + k_h * dilation_h;
                int w_in = w_out * stride_w - padding_w + k_w * dilation_w;
                if (h_in >= 0 && h_in < H_in && w_in >= 0 && w_in < W_in) {
                    int weight_idx = (((c_out * (C_in / groups) + (c_in - c_in_start)) * K_h + k_h) * K_w) + k_w;
                    shared_weight[threadIdx.y * blockDim.x + threadIdx.x] = weight[weight_idx];
                    value += shared_input[threadIdx.y * blockDim.x + threadIdx.x] * shared_weight[threadIdx.y * blockDim.x + threadIdx.x];
                }
            }
        }
        __syncthreads();
    }

    int output_idx = ((n * C_out + c_out) * H_out + h_out) * W_out + w_out;
    output[output_idx] = value;
}

torch::Tensor conv2d_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias_opt,
    std::vector<int64_t> stride,
    std::vector<int64_t> padding,
    std::vector<int64_t> dilation,
    int64_t groups
) {
    input = input.contiguous();
    weight = weight.contiguous();

    TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be on CUDA");

    if (bias_opt.has_value()) {
        TORCH_CHECK(bias_opt.value().is_cuda(), "Bias tensor must be on CUDA if provided");
    }

    int64_t N = input.size(0);
    int64_t C_in = input.size(1);
    int64_t H_in = input.size(2);
    int64_t W_in = input.size(3);
    int64_t C_out = weight.size(0);
    int64_t K_h = weight.size(2);
    int64_t K_w = weight.size(3);
    int64_t stride_h = stride[0];
    int64_t stride_w = stride[1];
    int64_t padding_h = padding[0];
    int64_t padding_w = padding[1];
    int64_t dilation_h = dilation[0];
    int64_t dilation_w = dilation[1];

    int64_t H_out = (H_in + 2 * padding_h - dilation_h * (K_h - 1) - 1) / stride_h + 1;
    int64_t W_out = (W_in + 2 * padding_w - dilation_w * (K_w - 1) - 1) / stride_w + 1;

    auto output = torch::zeros({N, C_out, H_out, W_out}, input.options());

    const float* input_ptr = input.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = nullptr;
    if (bias_opt.has_value()) {
        bias_ptr = bias_opt.value().contiguous().data_ptr<float>();
    }
    float* output_ptr = output.data_ptr<float>();

    dim3 threads(16, 16);
    dim3 blocks(
        (W_out + threads.x - 1) / threads.x,
        (H_out + threads.y - 1) / threads.y,
        N * C_out
    );

    size_t shared_mem_size = threads.x * threads.y * sizeof(float) * 2;

    conv2d_cuda_kernel<<<blocks, threads, shared_mem_size>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        N, C_in, H_in, W_in,
        C_out, H_out, W_out,
        K_h, K_w,
        stride_h, stride_w,
        padding_h, padding_w,
        dilation_h, dilation_w,
        groups
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &conv2d_cuda, "Custom 2D convolution (CUDA)",
        py::arg("input"),
        py::arg("weight"),
        py::arg("bias") = py::none(),
        py::arg("stride") = std::vector<int64_t>{1, 1},
        py::arg("padding") = std::vector<int64_t>{0, 0},
        py::arg("dilation") = std::vector<int64_t>{1, 1},
        py::arg("groups") = 1
    );
}