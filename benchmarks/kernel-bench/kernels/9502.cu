#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// Declare constant memory for weights and biases
__constant__ float const_weight[1024 * 64];  // 256KB
__constant__ float const_bias[1024];        // 4KB

__global__ void conv_transpose2d_forward_kernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch_size * out_channels * out_height * out_width;
  if (index >= total)
    return;

  // Decode index into (b, o, out_h, out_w)
  int w_out = index % out_width;
  int temp = index / out_width;
  int h_out = temp % out_height;
  temp /= out_height;
  int o = temp % out_channels;
  int b = temp / out_channels;

  float out_val = const_bias[o];  // Use constant memory for bias

  for (int c = 0; c < in_channels; ++c) {
    for (int p = 0; p < kernel_size; ++p) {
      int h_unscaled = h_out + padding - p * dilation;
      if (h_unscaled % stride != 0)
        continue;
      int h_in = h_unscaled / stride;
      if (h_in < 0 || h_in >= in_height)
        continue;
      for (int q = 0; q < kernel_size; ++q) {
        int w_unscaled = w_out + padding - q * dilation;
        if (w_unscaled % stride != 0)
          continue;
        int w_in = w_unscaled / stride;
        if (w_in < 0 || w_in >= in_width)
          continue;
        int input_idx = ((b * in_channels + c) * in_height + h_in) * in_width + w_in;
        int weight_idx = ((c * out_channels + o) * kernel_size + p) * kernel_size + q;
        out_val += input[input_idx] * const_weight[weight_idx];  // Use constant memory for weights
      }
    }
  }

  int output_idx = ((b * out_channels + o) * out_height + h_out) * out_width + w_out;
  output[output_idx] = out_val;
}

torch::Tensor conv_transpose2d_forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {
  
  int batch_size = input.size(0);
  int in_channels = input.size(1);
  int in_height = input.size(2);
  int in_width = input.size(3);
  int out_channels = weight.size(1);
  int kernel_size = weight.size(2);
  
  int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  int out_width  = (in_width - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  
  // Copy weight and bias to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(const_weight), weight.data_ptr<float>(), 
                     weight.numel() * sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(const_bias), bias.data_ptr<float>(), 
                     bias.numel() * sizeof(float));

  auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, 
                            input.options());
  
  int total_threads = batch_size * out_channels * out_height * out_width;
  int threads = 1024;
  int blocks = (total_threads + threads - 1) / threads;
  
  conv_transpose2d_forward_kernel<<<blocks, threads>>>(
      input.data_ptr<float>(),
      output.data_ptr<float>(),
      batch_size,
      in_channels,
      out_channels,
      in_height,
      in_width,
      kernel_size,
      out_height,
      out_width,
      stride,
      padding,
      dilation);
  
  return output;
}

torch::Tensor conv_transpose2d_forward_wrapper(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,
    int stride,
    int padding,
    int dilation) {
  
  int out_channels = weight.size(1);
  torch::Tensor bias;
  if (bias_obj.is(pybind11::none())) {
    bias = torch::zeros({out_channels}, weight.options());
  } else {
    bias = bias_obj.cast<torch::Tensor>();
  }
  
  return conv_transpose2d_forward_cuda(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &conv_transpose2d_forward_wrapper,
        "ConvTranspose2d forward (CUDA)",
        pybind11::arg("input"),
        pybind11::arg("weight"),
        pybind11::arg("bias"),
        pybind11::arg("stride"),
        pybind11::arg("padding"),
        pybind11::arg("dilation"));
}