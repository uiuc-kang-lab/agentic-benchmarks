#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t, int KERNEL_SIZE>
__global__ void max_pool2d_unrolled_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int stride,
    const int padding,
    const int dilation
) {
    const int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= batch_size * channels * output_height * output_width) return;

    const int ow = output_idx % output_width;
    const int oh = (output_idx / output_width) % output_height;
    const int c = (output_idx / (output_width * output_height)) % channels;
    const int b = output_idx / (output_width * output_height * channels);

    const int input_batch_stride = channels * input_height * input_width;
    const int input_channel_stride = input_height * input_width;
    const int base_idx = b * input_batch_stride + c * input_channel_stride;
    
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    const int ih_start = oh * stride - padding;
    const int iw_start = ow * stride - padding;

    if constexpr (KERNEL_SIZE == 2) {
        #pragma unroll
        for (int kh = 0; kh < 2; kh++) {
            const int ih = ih_start + kh * dilation;
            if (ih >= 0 && ih < input_height) {
                const int row_idx = base_idx + ih * input_width;
                #pragma unroll
                for (int kw = 0; kw < 2; kw++) {
                    const int iw = iw_start + kw * dilation;
                    if (iw >= 0 && iw < input_width) {
                        max_val = max(max_val, __ldg(&input[row_idx + iw]));
                    }
                }
            }
        }
    } else if constexpr (KERNEL_SIZE == 3) {
        #pragma unroll
        for (int kh = 0; kh < 3; kh++) {
            const int ih = ih_start + kh * dilation;
            if (ih >= 0 && ih < input_height) {
                const int row_idx = base_idx + ih * input_width;
                #pragma unroll
                for (int kw = 0; kw < 3; kw++) {
                    const int iw = iw_start + kw * dilation;
                    if (iw >= 0 && iw < input_width) {
                        max_val = max(max_val, __ldg(&input[row_idx + iw]));
                    }
                }
            }
        }
    } else {
        for (int kh = 0; kh < KERNEL_SIZE; kh++) {
            const int ih = ih_start + kh * dilation;
            if (ih >= 0 && ih < input_height) {
                const int row_idx = base_idx + ih * input_width;
                #pragma unroll 4
                for (int kw = 0; kw < KERNEL_SIZE; kw++) {
                    const int iw = iw_start + kw * dilation;
                    if (iw >= 0 && iw < input_width) {
                        max_val = max(max_val, __ldg(&input[row_idx + iw]));
                    }
                }
            }
        }
    }

    output[output_idx] = max_val;
}

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int threads = 256;
    const int blocks = (batch_size * channels * output_height * output_width + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_pool2d_cuda_forward", ([&] {
        if (kernel_size == 2) {
            max_pool2d_unrolled_kernel<scalar_t, 2><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size, channels,
                input_height, input_width,
                output_height, output_width,
                stride, padding, dilation
            );
        } else if (kernel_size == 3) {
            max_pool2d_unrolled_kernel<scalar_t, 3><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size, channels,
                input_height, input_width,
                output_height, output_width,
                stride, padding, dilation
            );
        } else {
            max_pool2d_unrolled_kernel<scalar_t, -1><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size, channels,
                input_height, input_width,
                output_height, output_width,
                stride, padding, dilation
            );
        }
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA)");
}