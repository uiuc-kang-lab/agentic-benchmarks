#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

// Modular device function for computing the sigmoid
__device__ inline float compute_sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}

// Modular device function for computing the swish activation
__device__ inline float compute_swish(float x) {
    return x * compute_sigmoid(x);
}

// Optimized Swish kernel using grid-stride loop and modular functions
__global__ void efficient_swish_kernel(const float* __restrict__ x, float* __restrict__ y, int64_t n) {
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < n; idx += blockDim.x * gridDim.x) {
        y[idx] = compute_swish(x[idx]);
    }
}

// CUDA forward function that validates tensor is on device and launches the kernel
torch::Tensor efficient_swish_forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    auto y = torch::empty_like(x);
    int64_t n = x.numel();
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;
    
    efficient_swish_kernel<<<blocks, threads>>>(x.data_ptr<float>(), y.data_ptr<float>(), n);
    
    return y;
}

// Pybind11 binding to expose the CUDA function
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &efficient_swish_forward, "Efficient Swish activation forward pass (CUDA)");
}
