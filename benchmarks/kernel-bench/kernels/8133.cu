#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel with minimized warp divergence and refactored conditional logic

template <typename scalar_t>
__global__ void conv_transpose2d_kernel_min_divergence(
    const scalar_t* input,
    const scalar_t* weight,
    const scalar_t* bias,
    scalar_t* output,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int out_channels,
    const int kernel_h,
    const int kernel_w,
    const int stride,
    const int padding,
    const int output_padding,
    const int groups,
    const int dilation,
    const int out_height,
    const int out_width
) {
    const int total_elements = batch_size * out_channels * out_height * out_width;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int gridStride = blockDim.x * gridDim.x;

    for (; idx < total_elements; idx += gridStride) {
        int n = idx;
        const int ow = n % out_width;
        n /= out_width;
        const int oh = n % out_height;
        n /= out_height;
        const int oc = n % out_channels;
        n /= out_channels;
        const int b = n;

        const int out_channels_per_group = out_channels / groups;
        const int g = oc / out_channels_per_group;
        const int oc_group = oc % out_channels_per_group;
        const int in_channels_per_group = in_channels / groups;
        const int ic_start = g * in_channels_per_group;

        scalar_t val = (bias != nullptr) ? bias[oc] : static_cast<scalar_t>(0);

        for (int kh = 0; kh < kernel_h; ++kh) {
            const int h_in = (oh - kh * dilation + padding) / stride;

            // Checks are combined in a single logical statement to reduce divergence
            bool valid_h = (h_in >= 0 && h_in < in_height && (oh - kh * dilation + padding) % stride == 0);

            for (int kw = 0; kw < kernel_w; ++kw) {
                const int w_in = (ow - kw * dilation + padding) / stride;

                // Combined checks for the width dimension
                bool valid_w = (w_in >= 0 && w_in < in_width && (ow - kw * dilation + padding) % stride == 0);

                if (valid_h && valid_w) {
                    for (int ic = 0; ic < in_channels_per_group; ++ic) {
                        const scalar_t x_val = input[b * in_channels * in_height * in_width
                                                  + (ic_start + ic) * in_height * in_width
                                                  + h_in * in_width + w_in];

                        const scalar_t w_val = weight[(ic_start + ic) * (out_channels_per_group * kernel_h * kernel_w) +
                                                       oc_group * kernel_h * kernel_w +
                                                       kh * kernel_w + kw];
                        val += x_val * w_val;
                    }
                }
            }
        }

        output[idx] = val;
    }
}


torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int output_padding,
    int groups,
    int dilation = 1
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(x.dim() == 4, "Input must be 4D");
    TORCH_CHECK(weight.dim() == 4, "Weight must be 4D");

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);

    const int out_channels = weight.size(1) * groups;
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);

    if (bias.has_value() && bias->defined()) {
        TORCH_CHECK(bias->numel() == out_channels, "Bias must have out_channels elements");
        TORCH_CHECK(bias->device().is_cuda(), "Bias must be a CUDA tensor");
    }

    const int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_h - 1) + output_padding + 1;
    const int out_width = (in_width - 1) * stride - 2 * padding + dilation * (kernel_w - 1) + output_padding + 1;

    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, x.options());

    const int total_elements = output.numel();

    constexpr int BLOCK_SIZE = 256;
    const int blocks = (total_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "conv_transpose2d_cuda_min_divergence", ([&] {
        conv_transpose2d_kernel_min_divergence<scalar_t><<<blocks, BLOCK_SIZE>>>(
            x.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            (bias.has_value() && bias->defined()) ? bias->data_ptr<scalar_t>() : nullptr,
            output.data_ptr<scalar_t>(),
            batch_size,
            in_channels,
            in_height,
            in_width,
            out_channels,
            kernel_h,
            kernel_w,
            stride,
            padding,
            output_padding,
            groups,
            dilation,
            out_height,
            out_width
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Transposed 2D Convolution with Minimized Warp Divergence (CUDA)",
          py::arg("x"), py::arg("weight"), py::arg("bias") = py::none(),
          py::arg("stride"), py::arg("padding"), py::arg("output_padding"),
          py::arg("groups"), py::arg("dilation") = 1);
}
