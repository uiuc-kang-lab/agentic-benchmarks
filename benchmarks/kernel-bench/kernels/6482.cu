#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void mean_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t outer_size,
    int64_t dim_size,
    int64_t inner_size) {
    
    extern __shared__ char shared_mem[];
    scalar_t* shared_data = reinterpret_cast<scalar_t*>(shared_mem);
    
    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;
    const unsigned int output_idx = bid;
    
    if (output_idx >= outer_size * inner_size) return;
    
    const unsigned int outer_idx = output_idx / inner_size;
    const unsigned int inner_idx = output_idx % inner_size;
    const unsigned int input_offset = outer_idx * dim_size * inner_size + inner_idx;
    
    // Load and sum elements using __ldg for read-only cache
    scalar_t thread_sum = 0;
    #pragma unroll 4
    for (unsigned int i = tid; i < dim_size; i += blockDim.x) {
        thread_sum += __ldg(&input[input_offset + i * inner_size]);
    }
    
    // Store the sum in shared memory
    shared_data[tid] = thread_sum;
    __syncthreads();
    
    // Parallel reduction in shared memory with template unrolling
    if (blockDim.x >= 512) { if (tid < 256) { shared_data[tid] += shared_data[tid + 256]; } __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) { shared_data[tid] += shared_data[tid + 128]; } __syncthreads(); }
    if (blockDim.x >= 128) { if (tid < 64) { shared_data[tid] += shared_data[tid + 64]; } __syncthreads(); }
    
    // Last warp reduction (no sync needed)
    if (tid < 32) {
        volatile scalar_t* smem = shared_data;
        if (blockDim.x >= 64) smem[tid] += smem[tid + 32];
        if (blockDim.x >= 32) smem[tid] += smem[tid + 16];
        if (blockDim.x >= 16) smem[tid] += smem[tid + 8];
        if (blockDim.x >= 8) smem[tid] += smem[tid + 4];
        if (blockDim.x >= 4) smem[tid] += smem[tid + 2];
        if (blockDim.x >= 2) smem[tid] += smem[tid + 1];
    }
    
    // Write result
    if (tid == 0) {
        output[output_idx] = shared_data[0] / static_cast<scalar_t>(dim_size);
    }
}

torch::Tensor mean_reduce_cuda(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();
    
    auto sizes = input.sizes().vec();
    int64_t dim_size = sizes[dim];
    
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }
    
    sizes.erase(sizes.begin() + dim);
    auto output = torch::empty(sizes, input.options());
    
    const int threads = 256;
    const int blocks = outer_size * inner_size;
    const int shared_mem_size = threads * sizeof(float);
    
    AT_DISPATCH_FLOATING_TYPES(input.type(), "mean_reduce_cuda", ([&] {
        mean_reduce_kernel<scalar_t><<<blocks, threads, shared_mem_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            dim_size,
            inner_size
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &mean_reduce_cuda, "Mean reduction (CUDA)");
}
