#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

template <typename scalar_t>
__global__ void l2_norm_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ norms,
    const int C,
    const int stride_C,
    const int outer_stride) {

    const int vec_idx = blockIdx.x;
    const int grid_y = gridDim.y;
    const int base_offset = vec_idx * outer_stride;

    scalar_t partial_sum = 0;
    int start = blockIdx.y * blockDim.x + threadIdx.x;
    int stride = blockDim.x * grid_y;
    for (int i = start; i < C; i += stride) {
        scalar_t val = input[base_offset + i * stride_C];
        partial_sum += val * val;
    }

    // Warp-level reduction
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        partial_sum += __shfl_down_sync(0xffffffff, partial_sum, offset);
    }

    __shared__ scalar_t shared[32];
    int lane = threadIdx.x & (warpSize - 1);
    int warp_id = threadIdx.x / warpSize;
    if (lane == 0) {
        shared[warp_id] = partial_sum;
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        scalar_t block_sum = 0;
        int num_warps = (blockDim.x + warpSize - 1) / warpSize;
        for (int i = 0; i < num_warps; i++) {
            block_sum += shared[i];
        }
        atomicAdd(&norms[vec_idx], block_sum);
    }
}

template <typename scalar_t>
__global__ void l2_norm_normalize_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ norms,
    const int C,
    const int stride_C,
    const int outer_stride) {

    const int vec_idx = blockIdx.x;
    const int base_offset = vec_idx * outer_stride;

    scalar_t norm_val = sqrt(norms[vec_idx]) + static_cast<scalar_t>(1e-12);
    scalar_t inv_norm = static_cast<scalar_t>(1.0) / norm_val;

    int grid_y = gridDim.y;
    int start = blockIdx.y * blockDim.x + threadIdx.x;
    int stride = blockDim.x * grid_y;
    for (int i = start; i < C; i += stride) {
        int index = base_offset + i * stride_C;
        output[index] = input[index] * inv_norm;
    }
}

torch::Tensor forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(input.dim() >= 2, "Input must have at least 2 dimensions");

    const int C = input.size(1);
    const int total_vectors = input.numel() / C;
    const int stride_C = input.stride(1);
    const int outer_stride = input.stride(0);

    auto output = torch::empty_like(input);
    auto norms = torch::zeros({total_vectors}, input.options());

    const int threads = 256;
    int blocksPerVector = (C + threads - 1) / threads;
    dim3 grid(total_vectors, blocksPerVector);  // Key change: Use full blocksPerVector

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "l2_norm_reduce", ([&] {
        l2_norm_reduce_kernel<scalar_t><<<grid, threads>>>(
            input.data_ptr<scalar_t>(),
            norms.data_ptr<scalar_t>(),
            C,
            stride_C,
            outer_stride
        );
    }));

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "l2_norm_normalize", ([&] {
        l2_norm_normalize_kernel<scalar_t><<<grid, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            norms.data_ptr<scalar_t>(),
            C,
            stride_C,
            outer_stride
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized L2 normalization with balanced grid");
}