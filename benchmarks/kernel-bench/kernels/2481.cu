#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t, int TILE_SIZE=32>
__global__ void matmul_transpose_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ C,
    const int M,
    const int N,
    const int K) {
    
    __shared__ scalar_t tileA[TILE_SIZE][TILE_SIZE];
    __shared__ scalar_t tileB[TILE_SIZE][TILE_SIZE];
    
    const int row = blockIdx.x * TILE_SIZE + threadIdx.x;
    const int col = blockIdx.y * TILE_SIZE + threadIdx.y;
    
    scalar_t sum = 0;
    const int numTiles = (K + TILE_SIZE - 1) / TILE_SIZE;
    
    for (int t = 0; t < numTiles; ++t) {
        const int k_offset = t * TILE_SIZE;
        
        if (k_offset + threadIdx.y < K && row < M) {
            tileA[threadIdx.y][threadIdx.x] = A[(k_offset + threadIdx.y) * M + row];
        } else {
            tileA[threadIdx.y][threadIdx.x] = 0.0;
        }
        
        if (k_offset + threadIdx.x < K && col < N) {
            tileB[threadIdx.y][threadIdx.x] = B[col * K + k_offset + threadIdx.x];
        } else {
            tileB[threadIdx.y][threadIdx.x] = 0.0;
        }
        
        __syncthreads();
        
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum = fma(tileA[k][threadIdx.x], tileB[threadIdx.y][k], sum);
        }
        
        if (t < numTiles - 1) {
            __syncthreads();
        }
    }
    
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

torch::Tensor matmul_transpose_cuda(torch::Tensor A, torch::Tensor B) {
    const int K = A.size(0);
    const int M = A.size(1);
    const int N = B.size(0);
    
    auto C = torch::empty({M, N}, A.options());
    
    constexpr int TILE_SIZE = 32;
    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((M + TILE_SIZE - 1) / TILE_SIZE,
                (N + TILE_SIZE - 1) / TILE_SIZE);
    
    AT_DISPATCH_FLOATING_TYPES(A.type(), "matmul_transpose_kernel", ([&] {
        matmul_transpose_kernel<scalar_t, TILE_SIZE><<<blocks, threads>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, N, K
        );
    }));
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_transpose_cuda, "Optimized matrix multiplication with transpose (CUDA)");
}