#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>

// Combined kernel that minimizes warp divergence by precomputing valid ranges (kernel2) and
// uses loop unrolling for common kernel sizes (kernel1) to improve performance.

template <typename scalar_t>
__global__ void max_pool2d_combined_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation
) {
    // Use 1D indexing for all output elements
    int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_outputs = batch_size * channels * output_height * output_width;
    if (output_idx >= total_outputs) return;

    // Calculate indices for output: ow, oh, channel and batch
    int ow = output_idx % output_width;
    int oh = (output_idx / output_width) % output_height;
    int c  = (output_idx / (output_width * output_height)) % channels;
    int b  = output_idx / (output_width * output_height * channels);

    // Compute the top-left corner in the input corresponding to this output element
    int base_h = oh * stride - padding;
    int base_w = ow * stride - padding;

    // Offset to the beginning of the corresponding input channel
    int input_channel_offset = (b * channels + c) * input_height * input_width;

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();

    // For common kernel sizes, use unrolled loops for efficiency
    if (kernel_size == 2) {
        #pragma unroll
        for (int kh = 0; kh < 2; kh++) {
            #pragma unroll
            for (int kw = 0; kw < 2; kw++) {
                int ih = base_h + kh * dilation;
                int iw = base_w + kw * dilation;
                if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
                    int input_idx = input_channel_offset + ih * input_width + iw;
                    max_val = max(max_val, input[input_idx]);
                }
            }
        }
    } else if (kernel_size == 3) {
        #pragma unroll
        for (int kh = 0; kh < 3; kh++) {
            #pragma unroll
            for (int kw = 0; kw < 3; kw++) {
                int ih = base_h + kh * dilation;
                int iw = base_w + kw * dilation;
                if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
                    int input_idx = input_channel_offset + ih * input_width + iw;
                    max_val = max(max_val, input[input_idx]);
                }
            }
        }
    } else {
        // Precompute valid kernel index ranges to minimize divergence
        int kh_start = 0;
        if (base_h < 0) {
            kh_start = (-base_h + dilation - 1) / dilation;
        }
        int kh_end = kernel_size;
        if (base_h + (kernel_size - 1) * dilation >= input_height) {
            int possible_end = (input_height - base_h + dilation - 1) / dilation;
            kh_end = possible_end < kernel_size ? possible_end : kernel_size;
        }

        int kw_start = 0;
        if (base_w < 0) {
            kw_start = (-base_w + dilation - 1) / dilation;
        }
        int kw_end = kernel_size;
        if (base_w + (kernel_size - 1) * dilation >= input_width) {
            int possible_end = (input_width - base_w + dilation - 1) / dilation;
            kw_end = possible_end < kernel_size ? possible_end : kernel_size;
        }

        for (int kh = kh_start; kh < kh_end; ++kh) {
            int ih = base_h + kh * dilation;
            for (int kw = kw_start; kw < kw_end; ++kw) {
                int iw = base_w + kw * dilation;
                int input_idx = input_channel_offset + ih * input_width + iw;
                max_val = max(max_val, input[input_idx]);
            }
        }
    }

    output[output_idx] = max_val;
}


// Forward function that sets up kernel launch

torch::Tensor max_pool2d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);

    const auto output_height = ((input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;
    const auto output_width = ((input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    const int num_outputs = batch_size * channels * output_height * output_width;
    const int threads = 256;
    const int blocks = (num_outputs + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool2d_cuda_forward", ([&] {
        max_pool2d_combined_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            channels,
            input_height,
            input_width,
            output_height,
            output_width,
            kernel_size,
            stride,
            padding,
            dilation
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_cuda_forward, "Max Pool 2D forward (CUDA) combined kernel");
}
