#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel that applies manual loop unrolling to speed up the reverse cumulative sum operation
// This kernel performs the reverse operation using warp shuffles and shared memory to compute offsets.

template <typename scalar_t>
__global__ void unrolled_reverse_cumsum_kernel(const scalar_t* __restrict__ input,
                                               scalar_t* __restrict__ output,
                                               int64_t n) {
    int row = blockIdx.x;
    const int64_t row_offset = row * n;

    int tid = threadIdx.x;
    int lane = tid & 31;  // Lane index within the warp

    // Load element in reverse order
    scalar_t val = 0;
    if (tid < n) {
        val = input[row_offset + (n - 1 - tid)];
    }

    // Perform warp-level inclusive scan using shuffle intrinsics with unrolled loop
    #pragma unroll
    for (int offset = 1; offset < 32; offset *= 2) {
        scalar_t tmp = __shfl_up_sync(0xffffffff, val, offset);
        if (lane >= offset) {
            val += tmp;
        }
    }

    // Each warp's last active thread writes its result to shared memory
    __shared__ scalar_t warp_sums[32];
    int warp_id = tid >> 5;
    if (tid < n && (lane == 31 || tid == n - 1)) {
        warp_sums[warp_id] = val;
    }
    __syncthreads();

    // Each thread computes the offset for its warp by summing the totals of previous warps
    if (tid < n) {
        scalar_t warp_offset = 0;
        for (int w = 0; w < warp_id; w++) {
            warp_offset += warp_sums[w];
        }
        scalar_t final_val = val + warp_offset;
        // Write the computed cumulative sum back in the original order
        output[row_offset + (n - 1 - tid)] = final_val;
    }
}

at::Tensor reverse_cumsum(at::Tensor x, int64_t dim) {
    // Ensure the tensor is contiguous and on CUDA
    x = x.contiguous();
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    
    const int ndim = x.dim();
    TORCH_CHECK(dim >= 0 && dim < ndim, "dim out of range");

    auto output = at::empty_like(x);

    int64_t n = x.size(dim);
    int64_t outer = x.numel() / n;

    // Use the optimized kernel only if the cum-sum is performed along the last dimension and n is not large
    if (dim == ndim - 1 && n <= 1024) {
        // Determine number of threads as next power of 2 >= n (capped at 1024)
        int threads = 1;
        while (threads < n) {
            threads *= 2;
        }
        if (threads > 1024) threads = 1024;

        dim3 blocks(outer);
        dim3 threadsPerBlock(threads);

        AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "unrolled_reverse_cumsum_kernel", ([&] {
            unrolled_reverse_cumsum_kernel<scalar_t><<<blocks, threadsPerBlock>>>(
                x.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                n);
        }));
    } else {
        // Fallback to the flip-cumsum-flip method for non-ideal cases
        auto x_flipped = x.flip(dim);
        auto cumsum = x_flipped.cumsum(dim);
        output = cumsum.flip(dim);
    }

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &reverse_cumsum, "Reverse cumulative sum with unrolled loops (CUDA)");
}