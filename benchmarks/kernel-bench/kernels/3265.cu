#include "hip/hip_runtime.h"
#include <torch/extension.h>

// Optimized kernel using a grid-stride loop to process multiple elements per thread.
// No atomic operations are used as each thread works on independent output elements.

__global__ void optimized_swish_kernel(const float* __restrict__ x, float* __restrict__ y, int64_t n) {
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < n; idx += blockDim.x * gridDim.x) {
        float val = x[idx];
        float sigmoid = 1.0f / (1.0f + expf(-val));
        y[idx] = val * sigmoid;
    }
}

torch::Tensor optimized_swish_forward(torch::Tensor x) {
    TORCH_CHECK(x.is_cuda(), "Input tensor must be on CUDA");
    auto y = torch::empty_like(x);
    int64_t n = x.numel();

    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;

    optimized_swish_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        n
    );

    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &optimized_swish_forward, "Optimized 25_Swish activation forward pass (CUDA)");
}
