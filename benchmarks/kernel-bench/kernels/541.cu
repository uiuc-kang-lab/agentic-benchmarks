#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

template<typename T>
__device__ __forceinline__ T load_vector(const float* __restrict__ ptr) {
    return __ldg(reinterpret_cast<const T*>(ptr));
}

template<typename T>
__device__ __forceinline__ void store_vector(float* __restrict__ ptr, T val) {
    *reinterpret_cast<T*>(ptr) = val;
}

__device__ __forceinline__ float4 compute_multiplication(float4 input, float scalar) {
    float4 result;
    result.x = input.x * scalar;
    result.y = input.y * scalar;
    result.z = input.z * scalar;
    result.w = input.w * scalar;
    return result;
}

__device__ __forceinline__ void process_vector4(const float* __restrict__ A,
                                               float* __restrict__ C,
                                               float s,
                                               int idx) {
    float4 input = load_vector<float4>(&A[idx]);
    float4 result = compute_multiplication(input, s);
    store_vector<float4>(&C[idx], result);
}

__device__ __forceinline__ void process_scalar(const float* __restrict__ A,
                                             float* __restrict__ C,
                                             float s,
                                             int idx) {
    C[idx] = __ldg(&A[idx]) * s;
}

__global__ void multiplyKernelModular(const float* __restrict__ A,
                                     float* __restrict__ C,
                                     float s,
                                     int64_t size) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int vector_size = 4;
    
    // Calculate total number of vector4 elements to process
    const int num_vector4_elements = size / vector_size;
    
    // Process elements using float4
    for (int vid = tid; vid < num_vector4_elements; vid += stride) {
        process_vector4(A, C, s, vid * vector_size);
    }
    
    // Handle remaining elements - assign them to the first few threads only
    const int remaining_start = num_vector4_elements * vector_size;
    const int remaining = size - remaining_start;
    
    if (tid < remaining) {
        process_scalar(A, C, s, remaining_start + tid);
    }
}

torch::Tensor forward(torch::Tensor A, float s) {
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor.");
    TORCH_CHECK(A.scalar_type() == torch::kFloat, "Input tensor A must be of type float.");

    auto C = torch::empty_like(A);
    int64_t size = A.numel();
    
    const int threads = 256;
    const int vector_size = 4;
    const int blocks = std::min(256, static_cast<int>((size + threads * vector_size - 1) / (threads * vector_size)));

    multiplyKernelModular<<<blocks, threads>>>(A.data_ptr<float>(),
                                             C.data_ptr<float>(),
                                             s,
                                             size);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Modular optimized matrix-scalar multiplication kernel");
}