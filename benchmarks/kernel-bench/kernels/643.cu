#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 32

// CUDA kernel for matrix multiplication using shared memory tiling
// Experimenting with different block sizes
template <typename scalar_t>
__global__ void matmul_cuda_kernel(const scalar_t* __restrict__ A, const scalar_t* __restrict__ B,
                                   scalar_t* __restrict__ C, int M, int K, int N) {
    __shared__ scalar_t sA[TILE_WIDTH][TILE_WIDTH];
    __shared__ scalar_t sB[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    scalar_t value = 0;

    // Loop over tiles
    for (int t = 0; t < (K + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {
        // Load elements into shared memory
        if (row < M && t * TILE_WIDTH + threadIdx.x < K)
            sA[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_WIDTH + threadIdx.x];
        else
            sA[threadIdx.y][threadIdx.x] = 0;

        if (t * TILE_WIDTH + threadIdx.y < K && col < N)
            sB[threadIdx.y][threadIdx.x] = B[(t * TILE_WIDTH + threadIdx.y) * N + col];
        else
            sB[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();

        #pragma unroll
        for (int i = 0; i < TILE_WIDTH; ++i) {
            value += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        C[row * N + col] = value;
    }
}

// Forward function
torch::Tensor module_fn(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input tensor B must be a CUDA tensor");

    int64_t M = A.size(0);
    int64_t K = A.size(1);
    int64_t N = B.size(1);

    TORCH_CHECK(K == B.size(0), "Inner dimensions of A and B must match");

    auto C = torch::empty({M, N}, A.options());

    // Experiment with different block sizes
    dim3 threads_per_block(TILE_WIDTH, TILE_WIDTH);
    dim3 num_blocks((N + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "matmul_cuda_kernel", ([&] {
        matmul_cuda_kernel<scalar_t><<<num_blocks, threads_per_block>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, K, N);
    }));

    hipDeviceSynchronize();

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &module_fn, "Matrix multiplication forward (CUDA) with block size experimentation");
}
