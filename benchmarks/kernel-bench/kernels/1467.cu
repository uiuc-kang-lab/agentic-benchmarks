#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 64
#define THREAD_TILE 4
#define MAX_MATRIX_DIM 8192

// Constant memory for matrix dimensions and number of tiles
__constant__ int d_N;
__constant__ int d_num_tiles;

// This kernel uses vectorized 128-bit aligned loads (float4) along with the __ldg() intrinsic
// to optimize read-only global memory access. Each thread computes a 4x4 sub-tile of the output matrix.

__global__ void vec_ldg_aligned_matmul(const float* __restrict__ A,
                                         const float* __restrict__ B,
                                         float* __restrict__ C) {
    // Shared memory tiles for A and B
    __shared__ float s_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float s_B[BLOCK_SIZE][BLOCK_SIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;

    // The block computes a tile of size BLOCK_SIZE x BLOCK_SIZE in C
    // Each thread computes a 4x4 sub-tile. Thus, blockDim = (BLOCK_SIZE/THREAD_TILE, BLOCK_SIZE/THREAD_TILE) = (16, 16).
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Compute the starting global indices for the 4x4 output computed by this thread
    int rowStart = by * BLOCK_SIZE + ty * THREAD_TILE;
    int colStart = bx * BLOCK_SIZE + tx * THREAD_TILE;

    // Registers to accumulate a 4x4 sub-tile
    float regC[THREAD_TILE][THREAD_TILE] = { {0.f, 0.f, 0.f, 0.f},
                                               {0.f, 0.f, 0.f, 0.f},
                                               {0.f, 0.f, 0.f, 0.f},
                                               {0.f, 0.f, 0.f, 0.f} };

    // Loop over tiles in the k-dimension
    for (int t = 0; t < d_num_tiles; t++) {
        // Load tile from A into shared memory using vectorized loads.
        int total_A_loads = (BLOCK_SIZE * BLOCK_SIZE) / 4;
        int tid = ty * blockDim.x + tx;

        for (int i = tid; i < total_A_loads; i += (blockDim.x * blockDim.y)) {
            int a_row_in_tile = i / (BLOCK_SIZE / 4);
            int a_col_group = i % (BLOCK_SIZE / 4);

            int a_global_row = by * BLOCK_SIZE + a_row_in_tile;
            int a_global_col = t * BLOCK_SIZE + a_col_group * 4;

            float4 A_vec;
            if (a_global_row < d_N && (a_global_col + 3) < d_N) {
                const float4* A_vec_ptr = reinterpret_cast<const float4*>(A);
                int index = a_global_row * d_N + a_global_col;
                A_vec = __ldg(&A_vec_ptr[index / 4]);
            } else {
                float tmp[4] = {0.f, 0.f, 0.f, 0.f};
                for (int j = 0; j < 4; j++) {
                    int col = a_global_col + j;
                    if (a_global_row < d_N && col < d_N)
                        tmp[j] = __ldg(&A[a_global_row * d_N + col]);
                }
                A_vec.x = tmp[0];
                A_vec.y = tmp[1];
                A_vec.z = tmp[2];
                A_vec.w = tmp[3];
            }
            
            int dest_col = a_col_group * 4;
            s_A[a_row_in_tile][dest_col + 0] = A_vec.x;
            s_A[a_row_in_tile][dest_col + 1] = A_vec.y;
            s_A[a_row_in_tile][dest_col + 2] = A_vec.z;
            s_A[a_row_in_tile][dest_col + 3] = A_vec.w;
        }

        // Load tile from B into shared memory using vectorized loads
        int total_B_loads = (BLOCK_SIZE * BLOCK_SIZE) / 4;
        for (int i = tid; i < total_B_loads; i += (blockDim.x * blockDim.y)) {
            int b_row_in_tile = i / (BLOCK_SIZE / 4);
            int b_col_group = i % (BLOCK_SIZE / 4);
            int b_global_row = t * BLOCK_SIZE + b_row_in_tile;
            int b_global_col = bx * BLOCK_SIZE + b_col_group * 4;
            
            float4 B_vec;
            if (b_global_row < d_N && (b_global_col + 3) < d_N) {
                const float4* B_vec_ptr = reinterpret_cast<const float4*>(B);
                int index = b_global_row * d_N + b_global_col;
                B_vec = __ldg(&B_vec_ptr[index / 4]);
            } else {
                float tmp[4] = {0.f, 0.f, 0.f, 0.f};
                for (int j = 0; j < 4; j++) {
                    int col = b_global_col + j;
                    if (b_global_row < d_N && col < d_N)
                        tmp[j] = __ldg(&B[b_global_row * d_N + col]);
                }
                B_vec.x = tmp[0];
                B_vec.y = tmp[1];
                B_vec.z = tmp[2];
                B_vec.w = tmp[3];
            }
            int dest_col = b_col_group * 4;
            s_B[b_row_in_tile][dest_col + 0] = B_vec.x;
            s_B[b_row_in_tile][dest_col + 1] = B_vec.y;
            s_B[b_row_in_tile][dest_col + 2] = B_vec.z;
            s_B[b_row_in_tile][dest_col + 3] = B_vec.w;
        }

        __syncthreads();
        
        // Multiply the loaded tiles; each thread computes its 4x4 sub-block.
        int a_sub_row = ty * THREAD_TILE; // starting row in s_A for this thread
        int b_sub_col = tx * THREAD_TILE;   // starting col in s_B for this thread

        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; k++) {
            float a0 = s_A[a_sub_row + 0][k];
            float a1 = s_A[a_sub_row + 1][k];
            float a2 = s_A[a_sub_row + 2][k];
            float a3 = s_A[a_sub_row + 3][k];

            float b0 = s_B[k][b_sub_col + 0];
            float b1 = s_B[k][b_sub_col + 1];
            float b2 = s_B[k][b_sub_col + 2];
            float b3 = s_B[k][b_sub_col + 3];

            regC[0][0] += a0 * b0;
            regC[0][1] += a0 * b1;
            regC[0][2] += a0 * b2;
            regC[0][3] += a0 * b3;

            regC[1][0] += a1 * b0;
            regC[1][1] += a1 * b1;
            regC[1][2] += a1 * b2;
            regC[1][3] += a1 * b3;

            regC[2][0] += a2 * b0;
            regC[2][1] += a2 * b1;
            regC[2][2] += a2 * b2;
            regC[2][3] += a2 * b3;

            regC[3][0] += a3 * b0;
            regC[3][1] += a3 * b1;
            regC[3][2] += a3 * b2;
            regC[3][3] += a3 * b3;
        }

        __syncthreads();
    }

    // Write the 4x4 sub-tile from registers back to global memory C using vectorized stores if possible
    for (int i = 0; i < THREAD_TILE; i++) {
        int global_row = rowStart + i;
        if (global_row < d_N) {
            int global_col = colStart;
            if (global_col + 3 < d_N) {
                float4 out_val;
                out_val.x = regC[i][0];
                out_val.y = regC[i][1];
                out_val.z = regC[i][2];
                out_val.w = regC[i][3];
                float4* C_vec_ptr = reinterpret_cast<float4*>(C);
                int index = global_row * d_N + global_col;
                C_vec_ptr[index / 4] = out_val;
            } else {
                for (int j = 0; j < THREAD_TILE; j++) {
                    int global_col_j = global_col + j;
                    if (global_col_j < d_N)
                        C[global_row * d_N + global_col_j] = regC[i][j];
                }
            }
        }
    }
}

// C++ interface using Pybind11

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D matrices");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must have the same dimensions");
    TORCH_CHECK(A.size(0) <= MAX_MATRIX_DIM, "Matrix dimension exceeds maximum supported size");

    int N = A.size(0);
    int num_tiles = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipMemcpyToSymbol(HIP_SYMBOL(d_N), &N, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_tiles), &num_tiles, sizeof(int));

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, A.get_device());
    auto C = torch::zeros({N, N}, options);

    // Launch configuration: blockDim = (BLOCK_SIZE/THREAD_TILE, BLOCK_SIZE/THREAD_TILE) = (16, 16)
    dim3 threads(BLOCK_SIZE / THREAD_TILE, BLOCK_SIZE / THREAD_TILE);
    dim3 blocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    vec_ldg_aligned_matmul<<<blocks, threads>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>());

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Vectorized 128-bit Aligned Matrix Multiplication with __ldg (CUDA)");
}
