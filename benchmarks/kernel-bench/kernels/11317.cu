#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void cosine_similarity_loss_kernel_stride(const float* __restrict__ predictions,
                                                     const float* __restrict__ targets,
                                                     float* output,
                                                     int N,
                                                     int D) {
    extern __shared__ float s_data[];
    int row = blockIdx.x;
    int tid = threadIdx.x;
    int stride = blockDim.x;

    const float* pred_row = predictions + row * D;
    const float* target_row = targets + row * D;

    float sum_dot = 0.0f;
    float sum_pred_sq = 0.0f;
    float sum_target_sq = 0.0f;

    for (int i = tid; i < D; i += stride) {
        float p = pred_row[i];
        float t = target_row[i];
        sum_dot += p * t;
        sum_pred_sq += p * p;
        sum_target_sq += t * t;
    }

    float* s_dot = s_data;
    float* s_pred_sq = s_data + blockDim.x;
    float* s_target_sq = s_pred_sq + blockDim.x;

    s_dot[tid] = sum_dot;
    s_pred_sq[tid] = sum_pred_sq;
    s_target_sq[tid] = sum_target_sq;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            s_dot[tid] += s_dot[tid + s];
            s_pred_sq[tid] += s_pred_sq[tid + s];
            s_target_sq[tid] += s_target_sq[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        const float eps = 1e-8f;
        float norm_pred = sqrtf(s_pred_sq[0]);
        float norm_target = sqrtf(s_target_sq[0]);
        float denominator = norm_pred * norm_target;
        denominator = fmaxf(denominator, eps);
        
        float cos_sim = s_dot[0] / denominator;
        atomicAdd(output, 1.0f - cos_sim);
    }
}

torch::Tensor cosine_similarity_loss_forward_stride(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.dim() == 2, "predictions must be 2D");
    TORCH_CHECK(targets.dim() == 2, "targets must be 2D");
    TORCH_CHECK(predictions.sizes() == targets.sizes(), "Input tensors must have the same shape");
    TORCH_CHECK(predictions.scalar_type() == torch::kFloat32, "predictions must be float32");
    TORCH_CHECK(targets.scalar_type() == torch::kFloat32, "targets must be float32");

    int N = predictions.size(0);
    int D = predictions.size(1);

    auto output = torch::zeros({1}, predictions.options());
    const int block_size = 256;
    size_t shared_mem = 3 * block_size * sizeof(float);

    cosine_similarity_loss_kernel_stride<<<N, block_size, shared_mem>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        N,
        D
    );

    output.div_(N);
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward_stride", &cosine_similarity_loss_forward_stride, "Cosine Similarity Loss Forward with Stride (CUDA)");
}