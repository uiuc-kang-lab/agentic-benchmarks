#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef TILE_SIZE
#define TILE_SIZE 32
#endif

// Kernel to compute a chunk of rows of C using atomic operations where necessary.
__global__ void triangular_mm_kernel_atomic(const float* __restrict__ A,
                                             const float* __restrict__ B,
                                             float* __restrict__ C,
                                             int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            float sum = 0.f;
            for (int k = col; k <= row; ++k) {
                sum += A[row * N + k] * B[k * N + col];
            }
            atomicAdd(&C[row * N + col], sum);
        }
    }
}

// Forward function to handle setup and execute kernel.
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    triangular_mm_kernel_atomic<<<grid, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized triangular matrix multiplication using atomic operations (CUDA)");
}
