#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

template<typename T>
struct VectorType {};

template<>
struct VectorType<float> {
    using type = float4;
    static constexpr int size = 4;
};

__global__ void cumsum_vec4_kernel(const float* __restrict__ input,
                                   float* __restrict__ output,
                                   int outer_size,
                                   int inner_size,
                                   int stride) {
    using vec_t = VectorType<float>::type;
    constexpr int vec_size = VectorType<float>::size;
    
    int outer_idx = blockIdx.x;
    int inner_idx = threadIdx.x * vec_size;
    
    if (outer_idx < outer_size && inner_idx < inner_size) {
        int base = outer_idx * stride * inner_size;
        
        // Process each element in the inner dimension independently
        for (int v = 0; v < vec_size && inner_idx + v < inner_size; ++v) {
            float running_sum = 0.0f;
            for (int i = 0; i < stride; ++i) {
                int idx = base + i * inner_size + inner_idx + v;
                running_sum += input[idx];
                output[idx] = running_sum;
            }
        }
    }
}

torch::Tensor forward(torch::Tensor x, int dim) {
    CHECK_INPUT(x);

    auto output = torch::empty_like(x);
    int ndim = x.dim();
    dim = (dim + ndim) % ndim;

    int outer_size = 1;
    for (int i = 0; i < dim; ++i) outer_size *= x.size(i);

    int inner_size = 1;
    for (int i = dim + 1; i < ndim; ++i) inner_size *= x.size(i);

    int stride = x.size(dim);
    
    dim3 blocks(outer_size);
    dim3 threads((inner_size + VectorType<float>::size - 1) / VectorType<float>::size);
    
    cumsum_vec4_kernel<<<blocks, threads>>>(x.data_ptr<float>(),
                                          output.data_ptr<float>(),
                                          outer_size,
                                          inner_size,
                                          stride);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Vectorized CUDA cumulative sum");
}