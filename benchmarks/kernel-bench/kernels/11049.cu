#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel: each warp computes the loss for one sample using warp-level shuffle reductions
__global__ void cross_entropy_loss_kernel_warp_shfl(
    const float* __restrict__ logits,
    const int64_t* __restrict__ targets,
    float* __restrict__ losses,
    int batch_size,
    int num_classes
) {
    // Each warp processes one sample. Compute global warp id and lane id
    int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = global_thread_id / 32;  // assume warp size is 32
    int lane = global_thread_id % 32;

    // Ensure this warp corresponds to a valid sample
    if (warp_id >= batch_size) return;

    // Pointer to the logits for this sample
    const float* sample_logits = logits + warp_id * num_classes;
    int target_class = targets[warp_id];

    // Step 1: Compute the maximum logit using warp-level reduction
    float local_max = -1e38f; // a very small number
    for (int j = lane; j < num_classes; j += 32) {
        local_max = fmaxf(local_max, sample_logits[j]);
    }

    unsigned int mask = 0xFFFFFFFF;
    // Reduce within the warp to obtain the maximum
    for (int offset = 16; offset > 0; offset /= 2) {
        float other = __shfl_down_sync(mask, local_max, offset);
        local_max = fmaxf(local_max, other);
    }
    // Broadcast the maximum value to all lanes in the warp
    float max_val = __shfl_sync(mask, local_max, 0);

    // Step 2: Compute the sum of exp(logits - max_val) using warp-level reduction
    float local_sum = 0.0f;
    for (int j = lane; j < num_classes; j += 32) {
        local_sum += expf(sample_logits[j] - max_val);
    }
    for (int offset = 16; offset > 0; offset /= 2) {
        local_sum += __shfl_down_sync(mask, local_sum, offset);
    }
    float total_sum = __shfl_sync(mask, local_sum, 0);

    // Step 3: Retrieve the logit corresponding to the target class
    float target_logit = 0.0f;
    if (lane == 0) {
        target_logit = sample_logits[target_class];
    }
    target_logit = __shfl_sync(mask, target_logit, 0);

    // Step 4: Compute the loss for this sample. Only lane 0 writes the result.
    if (lane == 0) {
        losses[warp_id] = -(target_logit - max_val - logf(total_sum));
    }
}

// Forward function that wraps the kernel call
torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    TORCH_CHECK(predictions.is_cuda(), "predictions must be a CUDA tensor");
    TORCH_CHECK(targets.is_cuda(), "targets must be a CUDA tensor");
    
    TORCH_CHECK(predictions.dim() == 2, "predictions must be a 2D tensor");
    TORCH_CHECK(targets.dim() == 1, "targets must be a 1D tensor");
    
    TORCH_CHECK(predictions.dtype() == torch::kFloat32, "predictions must be Float32 tensor");
    TORCH_CHECK(targets.dtype() == torch::kInt64, "targets must be Int64 tensor");

    int batch_size = predictions.size(0);
    int num_classes = predictions.size(1);

    TORCH_CHECK(targets.size(0) == batch_size, "targets must have the same batch size as predictions");

    // Allocate output tensor for per-sample losses
    auto losses = torch::empty({batch_size}, predictions.options());

    // Launch configuration: each warp processes one sample
    int threads_per_block = 128; // Must be a multiple of 32
    int warps_per_block = threads_per_block / 32;
    int num_blocks = (batch_size + warps_per_block - 1) / warps_per_block;

    cross_entropy_loss_kernel_warp_shfl<<<num_blocks, threads_per_block>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<int64_t>(),
        losses.data_ptr<float>(),
        batch_size,
        num_classes
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Error in cross_entropy_loss_kernel_warp_shfl: ", hipGetErrorString(err));

    // Compute mean loss over the batch and return
    auto loss = losses.mean();
    return loss;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Cross Entropy Loss forward (CUDA) with warp-level shfl reduction");
}
