#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>
#include <cmath>

// Unrolled kernel using a compile-time block size parameter for performance tuning
// Supported block sizes: 32, 64, 128, 256, and 512

template <typename scalar_t, int BLOCK_SIZE>
__global__ void unroll_tuned_log_softmax_forward_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int dim_size) {

    // Each block handles one row (batch element)
    int batch_idx = blockIdx.x;
    const scalar_t* input_row = input + batch_idx * dim_size;
    scalar_t* output_row = output + batch_idx * dim_size;

    // Shared memory for reductions
    __shared__ scalar_t sdata[BLOCK_SIZE];

    // Phase 1: Compute the maximum value in the row
    scalar_t local_max = -std::numeric_limits<scalar_t>::infinity();
    for (int idx = threadIdx.x; idx < dim_size; idx += BLOCK_SIZE) {
        scalar_t val = input_row[idx];
        local_max = (val > local_max) ? val : local_max;
    }
    sdata[threadIdx.x] = local_max;
    __syncthreads();

    // Reduction to find max value
    #pragma unroll
    for (int stride = BLOCK_SIZE / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            sdata[threadIdx.x] = (sdata[threadIdx.x] > sdata[threadIdx.x + stride]) ? 
                                   sdata[threadIdx.x] : sdata[threadIdx.x + stride];
        }
        __syncthreads();
    }
    scalar_t max_val = sdata[0];
    __syncthreads();

    // Phase 2: Compute the sum of exp(x - max_val) for numerical stability
    scalar_t local_sum = 0;
    for (int idx = threadIdx.x; idx < dim_size; idx += BLOCK_SIZE) {
        // Compute exponentials
        scalar_t exp_val = exp(input_row[idx] - max_val);
        local_sum += exp_val;
    }
    sdata[threadIdx.x] = local_sum;
    __syncthreads();

    // Reduction to compute total sum
    #pragma unroll
    for (int stride = BLOCK_SIZE / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            sdata[threadIdx.x] += sdata[threadIdx.x + stride];
        }
        __syncthreads();
    }
    scalar_t sum = sdata[0];
    scalar_t log_sum = log(sum);
    __syncthreads();

    // Phase 3: Write back the final LogSoftmax values
    for (int idx = threadIdx.x; idx < dim_size; idx += BLOCK_SIZE) {
        output_row[idx] = (input_row[idx] - max_val) - log_sum;
    }
}


// Host function
// This function permutes the input so that the reduction occurs on the last dimension,
// selects an optimal block size from the set {32, 64, 128, 256, 512} based on dim_size,
// and then launches the tuned CUDA kernel.

torch::Tensor unroll_tuned_log_softmax_cuda_forward(torch::Tensor input, int64_t dim) {
    TORCH_CHECK(input.is_cuda(), "input must be a CUDA tensor");
    TORCH_CHECK(
        input.scalar_type() == torch::kFloat32 || input.scalar_type() == torch::kFloat64,
        "input must be float32 or float64");

    int64_t ndim = input.dim();
    TORCH_CHECK(dim >= -ndim && dim < ndim, "dim out of range");
    dim = dim >= 0 ? dim : dim + ndim;

    // Permute input so that the target dimension is the last dimension
    std::vector<int64_t> permute_dims;
    for (int64_t i = 0; i < ndim; ++i) {
        if (i != dim) {
            permute_dims.push_back(i);
        }
    }
    permute_dims.push_back(dim);

    input = input.permute(permute_dims).contiguous();
    int64_t batch_size = input.numel() / input.size(-1);
    int64_t dim_size = input.size(-1);

    auto output = torch::empty_like(input);

    // Select an optimal block size from {32, 64, 128, 256, 512}
    int optimal_block_size = 256; // Default value
    if (dim_size <= 32) {
        optimal_block_size = 32;
    } else if (dim_size <= 64) {
        optimal_block_size = 64;
    } else if (dim_size <= 128) {
        optimal_block_size = 128;
    } else if (dim_size <= 256) {
        optimal_block_size = 256;
    } else if (dim_size <= 512) {
        optimal_block_size = 512;
    } else {
        optimal_block_size = 512; // For larger dimensions, cap at 512 threads per block
    }

    const int blocks = batch_size;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "unroll_tuned_log_softmax_forward_cuda", ([&] {
        if (optimal_block_size == 32) {
            unroll_tuned_log_softmax_forward_kernel<scalar_t, 32><<<blocks, 32>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else if (optimal_block_size == 64) {
            unroll_tuned_log_softmax_forward_kernel<scalar_t, 64><<<blocks, 64>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else if (optimal_block_size == 128) {
            unroll_tuned_log_softmax_forward_kernel<scalar_t, 128><<<blocks, 128>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else if (optimal_block_size == 256) {
            unroll_tuned_log_softmax_forward_kernel<scalar_t, 256><<<blocks, 256>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        } else if (optimal_block_size == 512) {
            unroll_tuned_log_softmax_forward_kernel<scalar_t, 512><<<blocks, 512>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                dim_size);
        }
    }));

    // Inverse permutation to restore original data layout
    std::vector<int64_t> inverse_permute_dims(ndim);
    for (size_t i = 0; i < permute_dims.size(); ++i) {
        inverse_permute_dims[permute_dims[i]] = i;
    }
    output = output.permute(inverse_permute_dims);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &unroll_tuned_log_softmax_cuda_forward, "Unroll Tuned LogSoftmax forward (CUDA)");
}
