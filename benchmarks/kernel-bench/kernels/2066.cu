#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

// Define TILE_SIZE for tiling and BLOCK_SIZE for shared memory reduction.
#ifndef TILE_SIZE
#define TILE_SIZE 16
#endif

#define BLOCK_SIZE 128
#define WARP_SIZE 32

__global__ void hybrid_triangular_mm_kernel(const float* __restrict__ A,
                                             const float* __restrict__ B,
                                             float* __restrict__ C,
                                             int N) {
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    int row = blockRow * TILE_SIZE + threadIdx.y;
    int col = blockCol * TILE_SIZE + threadIdx.x;

    // Allocate shared memory for partial tiles of A and B
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    float sum = 0.0f;

    for (int t = 0; t < ceil(N / static_cast<float>(TILE_SIZE)); ++t) {
        if (row < N && t * TILE_SIZE + threadIdx.x < N && row >= col) {
            As[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && t * TILE_SIZE + threadIdx.y < N && row >= col) {
            Bs[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }
        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < N && col < N && row >= col) {
        C[row * N + col] = sum;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    hybrid_triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Hybrid Optimized Triangular Matrix Multiplication (CUDA)");
}
