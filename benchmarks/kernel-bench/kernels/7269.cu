#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define TILE_SIZE 16  // Using smaller tile size for better occupancy
#define KERNEL_SIZE 3
#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void conv2d_kernel(
    const float* input,
    const float* weight,
    const float* bias,
    float* output,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int out_channels,
    const int out_height,
    const int out_width,
    const int stride,
    const int padding) {
    
    __shared__ float shared_input[TILE_SIZE + KERNEL_SIZE - 1][TILE_SIZE + KERNEL_SIZE - 1];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x * TILE_SIZE;
    int by = blockIdx.y * TILE_SIZE;
    int b = blockIdx.z / out_channels;
    int oc = blockIdx.z % out_channels;
    
    float sum = bias ? bias[oc] : 0.0f;
    
    int start_h = by + ty - padding;
    int start_w = bx + tx - padding;

    for (int ic = 0; ic < in_channels; ic++) {
        float input_val = 0.0f;
        
        for (int kh = 0; kh < KERNEL_SIZE; kh++) {
            int h = start_h + kh;
            bool h_in_bounds = (h >= 0) && (h < in_height);

            for (int kw = 0; kw < KERNEL_SIZE; kw++) {
                int w = start_w + kw;
                bool w_in_bounds = (w >= 0) && (w < in_width);

                if (h_in_bounds && w_in_bounds) {
                    input_val = input[((b * in_channels + ic) * in_height + h) * in_width + w];
                } else {
                    input_val = 0.0f;
                }

                float weight_val = weight[((oc * in_channels + ic) * KERNEL_SIZE + kh) * KERNEL_SIZE + kw];
                sum += input_val * weight_val;
            }
        }
    }
    
    int out_h = by + ty;
    int out_w = bx + tx;
    if (out_h < out_height && out_w < out_width) {
        output[((b * out_channels + oc) * out_height + out_h) * out_width + out_w] = sum;
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int dilation,
    int groups) {
    
    CHECK_INPUT(x);
    CHECK_INPUT(weight);
    if (bias.has_value()) {
        CHECK_INPUT(bias.value());
    }
    
    auto batch_size = x.size(0);
    auto in_channels = x.size(1);
    auto in_height = x.size(2);
    auto in_width = x.size(3);
    auto out_channels = weight.size(0);
    auto out_height = (in_height + 2 * padding - dilation * (KERNEL_SIZE - 1) - 1) / stride + 1;
    auto out_width = (in_width + 2 * padding - dilation * (KERNEL_SIZE - 1) - 1) / stride + 1;
    
    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, x.options());
    
    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks(
        (out_width + TILE_SIZE - 1) / TILE_SIZE,
        (out_height + TILE_SIZE - 1) / TILE_SIZE,
        batch_size * out_channels
    );
    
    conv2d_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.has_value() ? bias.value().data_ptr<float>() : nullptr,
        output.data_ptr<float>(),
        batch_size,
        in_channels,
        in_height,
        in_width,
        out_channels,
        out_height,
        out_width,
        stride,
        padding
    );
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA forward function for 2D convolution");
}
