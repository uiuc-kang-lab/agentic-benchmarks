#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel that employs 2D grid-stride loops to evenly distribute work over both rows and columns
__global__ void diag_matmul_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    const int64_t N,
    const int64_t M
) {
    // Iterate over rows using grid-stride loop in the y-direction
    for (int row = blockIdx.y * blockDim.y + threadIdx.y; row < N; row += blockDim.y * gridDim.y) {
        // Load the diagonal element for this row once into a register
        float a_val = A[row];
        
        // Iterate over columns using grid-stride loop in the x-direction
        for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < M; col += blockDim.x * gridDim.x) {
            int idx = row * M + col;
            C[idx] = a_val * B[idx];
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.dim() == 1, "A must be a 1D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == B.size(0), "Dimension mismatch: A.size(0) must match B.size(0)");

    A = A.contiguous();
    B = B.contiguous();

    int64_t N = A.size(0);
    int64_t M = B.size(1);

    auto C = torch::empty({N, M}, B.options());

    // Define block dimensions (16x16 is a good balance) and grid dimensions computed to cover the entire matrix
    dim3 block(16, 16);
    dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    diag_matmul_kernel<<<grid, block>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N,
        M
    );

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Diagonal matrix multiplication using 2D grid-stride loops");
}
