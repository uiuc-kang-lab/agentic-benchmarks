#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>

// Device function to compute the partial sum of squares for each thread using strided access
__device__ inline float compute_partial_sum(const float* input, int numel, int idx, int stride) {
    float sum = 0.0f;
    for (int i = idx; i < numel; i += stride) {
        sum += input[idx] * input[idx];
    }
    return sum;
}

// Device function to perform block-level reduction using shared memory
__device__ inline void block_reduce(volatile float* shared_sum, int tid, int block_size) {
    for (int stride = block_size / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_sum[tid] += shared_sum[tid + stride];
        }
        __syncthreads();
    }
}

// CUDA kernel for computing sum of squares using modular device functions
__global__ void compute_norm_kernel(const float* input, float* norm_out, int numel) {
    __shared__ float shared_sum[256];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;
    int stride = blockDim.x * gridDim.x;

    // Each thread computes its partial sum
    float sum = compute_partial_sum(input, numel, idx, stride);
    shared_sum[tid] = sum;
    __syncthreads();

    // Reduce the partial sums within the block
    block_reduce(shared_sum, tid, blockDim.x);

    // Thread 0 aggregates the block result into the global norm using atomic addition
    if (tid == 0) {
        atomicAdd(norm_out, shared_sum[0]);
    }
}

// CUDA kernel for normalizing the tensor
__global__ void normalize_kernel(const float* input, float* output, float norm, int numel) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numel) {
        output[idx] = input[idx] / norm;
    }
}

// C++ forward function called from Python
torch::Tensor forward(torch::Tensor input) {
    // Validate input constraints
    TORCH_CHECK(input.is_cuda(), "Input tensor must be on GPU");
    TORCH_CHECK(input.is_contiguous(), "Input tensor must be contiguous");
    TORCH_CHECK(input.scalar_type() == torch::kFloat32, "Input must be float32");

    // Allocate output tensor and a tensor for the norm
    auto output = torch::empty_like(input);
    auto norm_tensor = torch::zeros({1}, input.options());

    // Raw pointers
    const float* input_ptr = input.data_ptr<float>();
    float* output_ptr = output.data_ptr<float>();
    float* norm_ptr = norm_tensor.data_ptr<float>();

    int numel = input.numel();
    const int threads = 256;
    const int blocks = min(65535, (numel + threads - 1) / threads);

    // Compute sum of squares using the modular kernel
    compute_norm_kernel<<<blocks, threads>>>(input_ptr, norm_ptr, numel);

    // Copy the computed norm sum from device to host and compute the square root
    float norm_val;
    hipMemcpy(&norm_val, norm_ptr, sizeof(float), hipMemcpyDeviceToHost);
    norm_val = sqrt(norm_val);

    // Normalize the tensor
    normalize_kernel<<<blocks, threads>>>(input_ptr, output_ptr, norm_val, numel);

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Modular Frobenius norm normalization");
}
