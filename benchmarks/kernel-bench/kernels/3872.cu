#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Define constant memory for frequently used data
__constant__ float constant_one = 1.0f;

__global__ void softsign_kernel_constant_memory(const float* x, float* out, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        out[idx] = x[idx] / (constant_one + fabsf(x[idx]));
    }
}

torch::Tensor forward(torch::Tensor x) {
    CHECK_INPUT(x);

    auto out = torch::empty_like(x);
    int num_elements = x.numel();
    int threads = 1024;
    int blocks = (num_elements + threads - 1) / threads;

    softsign_kernel_constant_memory<<<blocks, threads>>>(
        x.data_ptr<float>(), out.data_ptr<float>(), num_elements
    );

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Softsign activation with constant memory (CUDA)");}
