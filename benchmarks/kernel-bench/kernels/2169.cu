#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32

// Kernel using stride loops for efficient workload distribution
__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                       const float* __restrict__ B,
                                       float* __restrict__ C,
                                       const int N) {
    __shared__ float shA[TILE_SIZE][TILE_SIZE];
    __shared__ float shB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    if (row >= N || col >= N) return;
    if (row < col) {
        C[row * N + col] = 0.0f;
        return;
    }

    float sum = 0.0f;

    // Use stride loop to handle larger workloads
    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++) {
        int a_col = t * TILE_SIZE + threadIdx.x;
        if (a_col < N && a_col <= row)
            shA[threadIdx.y][threadIdx.x] = A[row * N + a_col];
        else
            shA[threadIdx.y][threadIdx.x] = 0.0f;

        int b_row = t * TILE_SIZE + threadIdx.y;
        if (b_row < N && b_row >= col)
            shB[threadIdx.y][threadIdx.x] = B[b_row * N + col];
        else
            shB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; k++) {
            if (t * TILE_SIZE + k < N && t * TILE_SIZE + k >= col && t * TILE_SIZE + k <= row) {
                sum += shA[threadIdx.y][k] * shB[k][threadIdx.x];
            }
        }

        __syncthreads();
    }

    C[row * N + col] = sum;
}

// C++ interface exposed to PyTorch
at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "A and B must be CUDA tensors");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D tensors");
    TORCH_CHECK(A.size(0) == A.size(1) && B.size(0) == B.size(1), "A and B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    const int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    triangular_mm_kernel<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA) with stride loop optimization");
}