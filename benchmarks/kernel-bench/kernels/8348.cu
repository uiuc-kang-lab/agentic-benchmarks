#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/util/Optional.h>

namespace py = pybind11;

__device__ inline bool compute_input_position(int out_pos, int k, int stride, int dilation, int padding, int L_in, int* in_pos) {
    *in_pos = out_pos * stride + k * dilation - padding;
    return (*in_pos >= 0) && (*in_pos < L_in);
}

__device__ inline int get_weight_index(int out_ch, int local_in_ch, int K, int group_size_in, int k) {
    return out_ch * (group_size_in * K) + local_in_ch * K + k;
}

__global__ void conv1d_forward_kernel(
    const float* __restrict__ x,
    const float* __restrict__ w,
    const float* __restrict__ bias,
    float* __restrict__ y,
    const int N,
    const int C_in,
    const int L_in,
    const int C_out,
    const int K,
    const int stride,
    const int padding,
    const int dilation,
    const int groups,
    const int L_out,
    const int group_size_in,
    const int group_size_out
) {
    // 2D grid/block mapping for coalesced memory access
    const int out_pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int out_ch = blockIdx.y * blockDim.y + threadIdx.y;
    const int n = blockIdx.z;

    if (n >= N || out_ch >= C_out || out_pos >= L_out) return;

    const int group_idx = out_ch / group_size_out;
    float val = 0.0f;

    #pragma unroll 4
    for (int local_in_ch = 0; local_in_ch < group_size_in; ++local_in_ch) {
        const int in_ch = group_idx * group_size_in + local_in_ch;
        
        #pragma unroll 4
        for (int k = 0; k < K; ++k) {
            int in_pos;
            if (compute_input_position(out_pos, k, stride, dilation, padding, L_in, &in_pos)) {
                const float x_val = __ldg(&x[n * C_in * L_in + in_ch * L_in + in_pos]);
                const float w_val = __ldg(&w[out_ch * (group_size_in * K) + local_in_ch * K + k]);
                val += x_val * w_val;
            }
        }
    }

    if (bias) val += __ldg(&bias[out_ch]);

    y[n * C_out * L_out + out_ch * L_out + out_pos] = val;
}

at::Tensor conv1d_forward_impl(
    const at::Tensor& x,
    const at::Tensor& weight,
    c10::optional<at::Tensor> bias_opt,
    int64_t stride,
    int64_t padding,
    int64_t dilation,
    int64_t groups
) {
    TORCH_CHECK(x.is_cuda(), "x must be CUDA tensor");
    TORCH_CHECK(weight.is_cuda(), "weight must be CUDA tensor");

    const int N = x.size(0);
    const int C_in = x.size(1);
    const int L_in = x.size(2);
    const int C_out = weight.size(0);
    const int K = weight.size(2);

    const int L_out = (L_in + 2 * padding - dilation * (K - 1) - 1) / stride + 1;
    TORCH_CHECK(L_out > 0, "Invalid output length");

    auto y = torch::empty({N, C_out, L_out}, x.options());
    const float* bias_ptr = bias_opt.has_value() ? bias_opt->data_ptr<float>() : nullptr;

    const int group_size_in = C_in / groups;
    const int group_size_out = C_out / groups;

    // 2D grid for output positions/channels, z dim for batches
    dim3 block(32, 4);  // 128 threads/block
    dim3 grid(
        (L_out + block.x - 1) / block.x,
        (C_out + block.y - 1) / block.y,
        N
    );

    conv1d_forward_kernel<<<grid, block>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias_ptr,
        y.data_ptr<float>(),
        N, C_in, L_in, C_out, K,
        (int)stride, (int)padding, (int)dilation, (int)groups, L_out,
        group_size_in, group_size_out
    );

    TORCH_CHECK(hipGetLastError() == hipSuccess, "Kernel launch failed");
    return y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward",
        [](at::Tensor x, at::Tensor weight, py::object bias,
           int64_t stride, int64_t padding, int64_t dilation, int64_t groups) {
            return conv1d_forward_impl(x, weight,
                bias.is_none() ? c10::nullopt : c10::optional<at::Tensor>(bias.cast<at::Tensor>()),
                stride, padding, dilation, groups);
        }, "Optimized 1D Conv with 2D grid mapping"
    );
}