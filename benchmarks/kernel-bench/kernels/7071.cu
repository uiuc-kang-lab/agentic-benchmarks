#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>
#include <c10/cuda/CUDAStream.h>

// Fused kernel using warp-level primitives for efficient min reduction
// along a specified dimension.
template <typename scalar_t>
__global__ void min_reduce_fused_warp_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int outer,
    const int r,
    const int inner) {

  const int warpSize = 32;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int warp_id = idx / warpSize;
  if (warp_id >= outer * inner) return;

  int lane = threadIdx.x % warpSize;
  int outer_idx = warp_id / inner;
  int inner_idx = warp_id % inner;

  int base = outer_idx * (r * inner) + inner_idx;

  // Initialize local_min to the maximum possible value
  scalar_t local_min = std::numeric_limits<scalar_t>::max();

  // Each thread in a warp processes different elements across 'r' dimension
  #pragma unroll
  for (int j = lane; j < r; j += warpSize) {
    scalar_t val = input[base + j * inner];
    local_min = (val < local_min) ? val : local_min;
  }

  // Reduce within the warp
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    scalar_t other = __shfl_down_sync(0xffffffff, local_min, offset);
    local_min = (other < local_min) ? other : local_min;
  }

  // First lane of each warp writes the result
  if (lane == 0) {
    output[warp_id] = local_min;
  }
}

// Forward function that translates shapes, dimensions and launches the kernel
torch::Tensor forward(torch::Tensor input, int64_t dim) {
  TORCH_CHECK(input.is_cuda(), "input must be a CUDA tensor");
  if (!input.is_contiguous()) {
    input = input.contiguous();
  }

  int ndim = input.dim();
  TORCH_CHECK(dim >= 0 && dim < ndim, "dim out of range");

  int outer = 1;
  for (int i = 0; i < dim; i++) {
    outer *= input.size(i);
  }
  int r = input.size(dim);
  int inner = 1;
  for (int i = dim + 1; i < ndim; i++) {
    inner *= input.size(i);
  }

  std::vector<int64_t> output_shape;
  for (int i = 0; i < ndim; i++) {
    if (i != dim) {
      output_shape.push_back(input.size(i));
    }
  }
  auto output = torch::empty(output_shape, input.options());

  int total_warps = outer * inner;
  const int threads_per_block = 128; // Optimized choice for balance of occupancy
  int num_blocks = (total_warps * 32 + threads_per_block - 1) / threads_per_block;

  AT_DISPATCH_ALL_TYPES(input.scalar_type(), "min_reduce_fused_warp_cuda", ([&] {
    min_reduce_fused_warp_kernel<scalar_t><<<num_blocks, threads_per_block, 0,
      c10::cuda::getCurrentCUDAStream().stream()>>>(
        input.data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>(),
        outer,
        r,
        inner);
  }));

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "Efficient min reduction using fused kernel and warp-level primitives (CUDA)");
}
