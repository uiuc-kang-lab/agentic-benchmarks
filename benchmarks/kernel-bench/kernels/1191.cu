#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

// This kernel computes a 4D tensor-matrix multiplication for a subset (chunk) of the batch dimension.
// A is of shape [BATCH_chunk, I, J, L] and B is of shape [L, K].
// Each thread computes one element of the output C of shape [BATCH_chunk, I, J, K].
__global__ void einsum_kernel_chunked(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int BATCH_chunk, int I, int J, int L, int K
) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = BATCH_chunk * I * J * K;
    if (global_idx >= total) return;

    // Decode the flat thread index into our 4D indices (for the batch chunk only)
    int k = global_idx % K;
    int tmp = global_idx / K;
    int j = tmp % J;
    tmp /= J;
    int i = tmp % I;
    int b_local = tmp / I;  // batch index within this chunk

    float sum = 0.0f;
    for (int l = 0; l < L; ++l) {
        int a_offset = b_local * I * J * L + i * J * L + j * L + l;
        int b_offset = l * K + k;
        sum += A[a_offset] * B[b_offset];
    }
    C[global_idx] = sum;
}

// The forward function splits the work along the batch dimension and uses multiple CUDA streams to
// overlap kernel execution with other device operations (including potential asynchronous memory
// transfers in a more complex pipeline). This pipelining can reduce total runtime on devices like
// the NVIDIA H100 GPU.

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(A.dim() == 4, "A must be 4D");
    TORCH_CHECK(B.dim() == 2, "B must be 2D");
    TORCH_CHECK(A.size(3) == B.size(0), "Dimension mismatch in l");

    int BATCH = A.size(0);
    int I = A.size(1);
    int J = A.size(2);
    int L = A.size(3);
    int K = B.size(1);

    // Allocate output tensor C on the same device and with the same options as A
    auto C = torch::zeros({BATCH, I, J, K}, A.options());

    // Decide on the number of streams. Here we choose 2 if there is more than one batch,
    // so that different chunks of the batch can be processed concurrently.
    int num_streams = (BATCH > 1) ? 2 : 1;
    int batch_chunk = (BATCH + num_streams - 1) / num_streams;  // ceiling division

    // Create CUDA streams
    std::vector<hipStream_t> streams(num_streams);
    for (int s = 0; s < num_streams; s++) {
        hipStreamCreate(&streams[s]);
    }

    // Launch kernels on each stream for a sub-batch of A and C
    for (int s = 0; s < num_streams; s++) {
        int start_batch = s * batch_chunk;
        int current_batch = std::min(batch_chunk, BATCH - start_batch);
        if (current_batch <= 0) break;

        // Offset the pointers so that the kernel sees a contiguous chunk of the batch dimension.
        const float* A_ptr = A.data_ptr<float>() + start_batch * I * J * L;
        float* C_ptr = C.data_ptr<float>() + start_batch * I * J * K;
        int total_elements = current_batch * I * J * K;

        int threads = 256;
        int blocks = (total_elements + threads - 1) / threads;

        einsum_kernel_chunked<<<blocks, threads, 0, streams[s]>>>(
            A_ptr, B.data_ptr<float>(), C_ptr,
            current_batch, I, J, L, K
        );
    }

    // Synchronize all streams to ensure that all kernels have finished execution
    for (int s = 0; s < num_streams; s++) {
        hipStreamSynchronize(streams[s]);
        hipStreamDestroy(streams[s]);
    }

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "4D tensor-matrix multiplication with async streams (CUDA)");
}
