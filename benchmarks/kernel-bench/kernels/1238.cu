#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

__global__ void einsum_kernel_3D_grid(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int BATCH, int I, int J, int L, int K
) {
    int batch_idx = blockIdx.z;
    int i_idx = blockIdx.y;
    int j_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (j_idx >= J || batch_idx >= BATCH || i_idx >= I) return;

    float sum = 0.0f;
    for(int l = 0; l < L; ++l) {
        int a_offset = batch_idx * I*J*L + i_idx*J*L + j_idx*L + l;
        int b_offset = l*K + threadIdx.x;
        sum += A[a_offset] * B[b_offset];
    }
    
    int c_offset = batch_idx * I*J*K + i_idx*J*K + j_idx*K + threadIdx.x;
    C[c_offset] = sum;
}

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(A.dim() == 4, "A must be 4D");
    TORCH_CHECK(B.dim() == 2, "B must be 2D");
    TORCH_CHECK(A.size(3) == B.size(0), "Dimension mismatch in l");

    int BATCH = A.size(0), I = A.size(1), J = A.size(2), L = A.size(3);
    int K = B.size(1);
    
    auto C = torch::zeros({BATCH, I, J, K}, A.options());
    
    dim3 threads(256);
    dim3 blocks((J + threads.x - 1) / threads.x, I, BATCH);
    
    einsum_kernel_3D_grid<<<blocks, threads>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        BATCH, I, J, L, K
    );
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "4D tensor-matrix multiplication with 3D grid (CUDA)");
}