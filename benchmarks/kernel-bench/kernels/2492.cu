#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t, int BLOCK_SIZE = 32>
__global__ void matmul_transpose_kernel(
    const scalar_t* __restrict__ A,
    const scalar_t* __restrict__ B,
    scalar_t* __restrict__ C,
    const int M,
    const int N,
    const int K) {
    
    const unsigned FULL_MASK = 0xffffffff;
    const int warp_size = 32;
    const int warp_id = threadIdx.y / (warp_size/BLOCK_SIZE);
    const int lane_id = threadIdx.x % warp_size;
    
    const int row = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    const int col = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    
    // Each thread computes one element of C
    scalar_t thread_sum = 0;
    
    // Process K dimension in chunks of warp_size
    for (int k_base = 0; k_base < K; k_base += warp_size) {
        scalar_t a_reg = 0, b_reg = 0;
        
        // Load values into registers
        if (k_base + lane_id < K) {
            if (row < M) {
                // Correct access pattern for transposed A
                a_reg = A[row * K + (k_base + lane_id)];
            }
            if (col < N) {
                b_reg = B[col * K + k_base + lane_id];
            }
        }
        
        // Perform warp-level multiplication and reduction
        #pragma unroll
        for (int offset = 0; offset < warp_size; ++offset) {
            scalar_t a_shifted = __shfl_sync(FULL_MASK, a_reg, offset);
            scalar_t b_shifted = __shfl_sync(FULL_MASK, b_reg, offset);
            thread_sum += a_shifted * b_shifted;
        }
    }
    
    // Write final result directly
    if (row < M && col < N) {
        C[row * N + col] = thread_sum;
    }
}

torch::Tensor matmul_transpose_cuda(torch::Tensor A, torch::Tensor B) {
    const int K = A.size(0);
    const int M = A.size(1);
    const int N = B.size(0);
    
    auto C = torch::empty({M, N}, A.options());
    
    constexpr int BLOCK_SIZE = 32;
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((M + BLOCK_SIZE - 1) / BLOCK_SIZE,
                (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    AT_DISPATCH_FLOATING_TYPES(A.type(), "matmul_transpose_kernel", ([&] {
        matmul_transpose_kernel<scalar_t, BLOCK_SIZE><<<blocks, threads>>>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            C.data_ptr<scalar_t>(),
            M, N, K
        );
    }));
    
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_transpose_cuda, "Warp-optimized matrix multiplication with transpose (CUDA)");
}