#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <type_traits>

// fast_clamp: minimize warp divergence by using intrinsic math functions
template <typename T>
__device__ __forceinline__ T fast_clamp(T x, T lo, T hi);

__device__ __forceinline__ float fast_clamp(float x, float lo, float hi) {
    return fminf(hi, fmaxf(lo, x));
}

__device__ __forceinline__ double fast_clamp(double x, double lo, double hi) {
    return fmin(hi, fmax(lo, x));
}


// This kernel evenly distributes workloads by precomputing the number of complete vectorized chunks
// and then assigning them uniformly among threads. It processes remaining tail elements separately.

template <typename scalar_t, int VEC_SIZE>
__global__ void hardsigmoid_kernel(const scalar_t* __restrict__ input,
                                   scalar_t* __restrict__ output,
                                   size_t numel) {
    // Each chunk holds VEC_SIZE elements
    size_t num_chunks = numel / VEC_SIZE;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int total_threads = blockDim.x * gridDim.x;

    constexpr scalar_t three = static_cast<scalar_t>(3.0);
    constexpr scalar_t sixth = static_cast<scalar_t>(1.0/6.0);

    // Select vector type based on precision: for float use float4, for double use double2
    using vec_t = typename std::conditional<
        std::is_same<scalar_t, float>::value,
        float4,
        double2
    >::type;

    // Evenly process full vectorized chunks
    for (size_t idx = tid; idx < num_chunks; idx += total_threads) {
        size_t base = idx * VEC_SIZE;
        vec_t chunk = *reinterpret_cast<const vec_t*>(&input[base]);
        scalar_t elems[VEC_SIZE];
        *reinterpret_cast<vec_t*>(elems) = chunk;

        #pragma unroll
        for (int i = 0; i < VEC_SIZE; i++) {
            scalar_t x = elems[i];
            x = (x + three) * sixth;  // computes (x + 3) / 6
            x = (x < static_cast<scalar_t>(0)) ? static_cast<scalar_t>(0) :
                (x > static_cast<scalar_t>(1) ? static_cast<scalar_t>(1) : x);
            elems[i] = x;
        }

        *reinterpret_cast<vec_t*>(&output[base]) = *reinterpret_cast<vec_t*>(elems);
    }

    // Process any remaining tail elements
    size_t tail_start = num_chunks * VEC_SIZE;
    for (size_t i = tail_start + tid; i < numel; i += total_threads) {
        scalar_t x = input[i];
        x = (x + three) * sixth;
        x = (x < static_cast<scalar_t>(0)) ? static_cast<scalar_t>(0) :
            (x > static_cast<scalar_t>(1) ? static_cast<scalar_t>(1) : x);
        output[i] = x;
    }
}

torch::Tensor forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be on CUDA");
    auto output = torch::empty_like(input);
    size_t numel = input.numel();

    const int threads = 256;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "evenly_distributed_hardsigmoid_cuda", ([&] {
        // Choose vector size depending on type: 4 for float (float4), 2 for double (double2)
        constexpr int vec_size = std::is_same<scalar_t, float>::value ? 4 : 2;
        size_t num_chunks = numel / vec_size;
        int blocks = (num_chunks + threads - 1) / threads;
        if (blocks == 0) {
            blocks = 1;
        }
        hardsigmoid_kernel<scalar_t, vec_size><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            numel);
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &forward, "Evenly Distributed HardSigmoid activation forward (CUDA)");
}
