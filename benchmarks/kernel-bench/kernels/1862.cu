#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   const int N) {
    const int warp_size = 32;
    const int lane_id = threadIdx.x % warp_size;
    const int warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / warp_size;
    const int row = warp_id;

    if (row < N) {
        // Each warp processes one row
        for (int col = lane_id; col < N; col += warp_size) {
            if (col <= row) {
                float sum = 0.0f;
                
                #pragma unroll 4
                for (int k = col; k <= row; k++) {
                    sum += A[row * N + k] * B[k * N + col];
                }
                
                C[row * N + col] = sum;
            } else {
                // Zero out upper triangular part
                C[row * N + col] = 0.0f;
            }
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    const int N = A.size(0);
    auto C = torch::empty_like(A);

    // Configure kernel launch parameters
    const int threads_per_block = 128;  // 4 warps per block
    const int warps_needed = N;
    const int blocks = (warps_needed * 32 + threads_per_block - 1) / threads_per_block;

    triangular_mm_kernel<<<blocks, threads_per_block>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}