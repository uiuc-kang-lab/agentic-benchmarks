#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>
#include <cmath>

__device__ __forceinline__ int compute_start_position(int out_idx, int stride, int padding) {
    return out_idx * stride - padding;
}

__device__ __forceinline__ int compute_pool_bounds(int start, int input_size, int kernel_size, int dilation, bool is_start) {
    if (is_start) {
        return (start < 0) ? ((-start + dilation - 1) / dilation) : 0;
    } else {
        int valid_max = (input_size - start + dilation - 1) / dilation;
        return min(kernel_size, valid_max);
    }
}

__device__ __forceinline__ int compute_input_index(
    int b, int c, int d, int h, int w,
    int channels, int input_d, int input_h, int input_w) {
    return (((b * channels + c) * input_d + d) * input_h + h) * input_w + w;
}

// Unified kernel using 1D grid configuration for better load balancing and coalesced memory access
// Combining the best practices from both kernels

template <typename scalar_t>
__global__ void optimized_maxpool3d_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t* __restrict__ indices,
    const int batch_size,
    const int channels,
    const int input_d, const int input_h, const int input_w,
    const int output_d, const int output_h, const int output_w,
    const int kernel_size,
    const int stride,
    const int padding,
    const int dilation) {

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size * channels * output_d * output_h * output_w) return;

    int w_out = idx % output_w;
    int h_out = (idx / output_w) % output_h;
    int d_out = (idx / (output_w * output_h)) % output_d;
    int c = (idx / (output_w * output_h * output_d)) % channels;
    int b = idx / (output_w * output_h * output_d * channels);

    const int d_start = compute_start_position(d_out, stride, padding);
    const int h_start = compute_start_position(h_out, stride, padding);
    const int w_start = compute_start_position(w_out, stride, padding);

    const int k_d_start = compute_pool_bounds(d_start, input_d, kernel_size, dilation, true);
    const int k_d_end = compute_pool_bounds(d_start, input_d, kernel_size, dilation, false);
    const int k_h_start = compute_pool_bounds(h_start, input_h, kernel_size, dilation, true);
    const int k_h_end = compute_pool_bounds(h_start, input_h, kernel_size, dilation, false);
    const int k_w_start = compute_pool_bounds(w_start, input_w, kernel_size, dilation, true);
    const int k_w_end = compute_pool_bounds(w_start, input_w, kernel_size, dilation, false);

    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    int max_index = -1;

    #pragma unroll
    for (int k_d = k_d_start; k_d < k_d_end; k_d++) {
        const int d_in = d_start + k_d * dilation;
        #pragma unroll
        for (int k_h = k_h_start; k_h < k_h_end; k_h++) {
            const int h_in = h_start + k_h * dilation;
            #pragma unroll
            for (int k_w = k_w_start; k_w < k_w_end; k_w++) {
                const int w_in = w_start + k_w * dilation;
                const int input_idx = compute_input_index(b, c, d_in, h_in, w_in,
                                                        channels, input_d, input_h, input_w);
                const scalar_t val = __ldg(&input[input_idx]);
                if (val > max_val) {
                    max_val = val;
                    max_index = input_idx;
                }
            }
        }
    }

    output[idx] = max_val;
    if (indices != nullptr) {
        indices[idx] = max_index;
    }
}

// Host function to launch the optimized kernel

torch::Tensor optimized_max_pool3d_cuda_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation,
    bool return_indices,
    bool ceil_mode) {

    auto input_sizes = input.sizes();
    const int batch_size = input_sizes[0];
    const int channels = input_sizes[1];
    const int input_d = input_sizes[2];
    const int input_h = input_sizes[3];
    const int input_w = input_sizes[4];

    const int output_d = ceil_mode ? 
        ceil((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_d + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    const int output_h = ceil_mode ?
        ceil((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_h + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);
    const int output_w = ceil_mode ?
        ceil((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1) :
        floor((input_w + 2 * padding - dilation * (kernel_size - 1) - 1) / float(stride) + 1);

    auto output = torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options());
    auto indices = return_indices ? 
        torch::empty({batch_size, channels, output_d, output_h, output_w}, input.options().dtype(torch::kLong)) :
        torch::Tensor();

    const int threads = 256;
    const int blocks = (batch_size * channels * output_d * output_h * output_w + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "optimized_max_pool3d_forward_cuda", ([&] {
        optimized_maxpool3d_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            return_indices ? indices.data_ptr<int64_t>() : nullptr,
            batch_size, channels,
            input_d, input_h, input_w,
            output_d, output_h, output_w,
            kernel_size, stride, padding, dilation);
    }));

    if (return_indices) {
        return torch::stack({output, indices}, 0);
    }
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &optimized_max_pool3d_cuda_forward, "Optimized Max Pool 3D forward (CUDA)");
}