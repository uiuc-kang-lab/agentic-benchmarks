#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void hinge_loss_strided_kernel(const float* __restrict__ predictions,
                                        const float* __restrict__ targets,
                                        float* __restrict__ output,
                                        const int n) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int num_threads = blockDim.x;
    const int grid_size = num_threads * gridDim.x;
    
    // Each thread processes multiple elements with stride
    for (int idx = bid * num_threads + tid; idx < n; idx += grid_size) {
        const float pred = __ldg(&predictions[idx]);
        const float target = __ldg(&targets[idx]);
        output[idx] = fmaxf(0.0f, 1.0f - pred * target);
    }
}

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    CHECK_INPUT(predictions);
    CHECK_INPUT(targets);

    int n = predictions.numel();
    torch::Tensor output = torch::empty_like(predictions);

    // Optimize thread and block count for better occupancy
    const int threads_per_block = 256;
    const int max_blocks = 256;  // Adjusted for better resource utilization
    const int num_blocks = min((n + threads_per_block - 1) / threads_per_block, max_blocks);

    hinge_loss_strided_kernel<<<num_blocks, threads_per_block>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    return torch::mean(output);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Strided Hinge Loss Forward");
}