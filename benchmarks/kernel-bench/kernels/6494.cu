#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Template function to reduce with optimized workload distribution
template <typename scalar_t>
__global__ void mean_reduce_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    int64_t outer_size,
    int64_t dim_size,
    int64_t inner_size) {
    
    extern __shared__ char shared_mem[];
    scalar_t* shared_data = reinterpret_cast<scalar_t*>(shared_mem);
    
    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;
    const unsigned int block_threads = blockDim.x;
    const unsigned int output_idx = bid;
    
    if (output_idx >= outer_size * inner_size) return;
    
    const unsigned int outer_idx = output_idx / inner_size;
    const unsigned int inner_idx = output_idx % inner_size;
    const unsigned int input_offset = outer_idx * dim_size * inner_size + inner_idx;
    
    // Load and sum elements with optimized workload distribution
    scalar_t thread_sum = 0;
    for (unsigned int i = tid; i < dim_size; i += block_threads) {
        if (!isnan(input[input_offset + i * inner_size])) {
        thread_sum += input[input_offset + i * inner_size];
    }
    }
    
    // Store the sum in shared memory
    shared_data[tid] = thread_sum;
    __syncthreads();
    
    // Parallel reduction in shared memory
    for (unsigned int stride = block_threads / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }
    
    // Write result
    if (tid == 0) {
        output[output_idx] = shared_data[0] / static_cast<scalar_t>(dim_size);
    }
}

// Host function to prepare and launch the CUDA kernel
torch::Tensor mean_reduce_cuda(torch::Tensor input, int64_t dim) {
    if (dim < 0) dim += input.dim();
    
    auto sizes = input.sizes().vec();
    int64_t dim_size = sizes[dim];
    
    int64_t outer_size = 1;
    for (int i = 0; i < dim; i++) {
        outer_size *= sizes[i];
    }
    
    int64_t inner_size = 1;
    for (int i = dim + 1; i < sizes.size(); i++) {
        inner_size *= sizes[i];
    }
    
    sizes.erase(sizes.begin() + dim);
    auto output = torch::empty(sizes, input.options());
    
    const int threads = 256;
    const int blocks = (outer_size * inner_size + threads - 1) / threads;
    const int shared_mem_size = threads * input.element_size();
    
    AT_DISPATCH_FLOATING_TYPES(input.type(), "mean_reduce_cuda", ([&] {
        mean_reduce_kernel<scalar_t><<<blocks, threads, shared_mem_size>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            outer_size,
            dim_size,
            inner_size
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &mean_reduce_cuda, "Mean reduction (CUDA)");
}
