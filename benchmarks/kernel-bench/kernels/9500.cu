#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <pybind11/pybind11.h>

// Device function to decode a linear index into (b, o, out_h, out_w)
__device__ inline void decode_index(int index, int out_width, int out_height, int out_channels,
                                      int &b, int &o, int &h_out, int &w_out) {
  w_out = index % out_width;
  int temp = index / out_width;
  h_out = temp % out_height;
  temp /= out_height;
  o = temp % out_channels;
  b = temp / out_channels;
}

// Device function to compute the convolution contribution for a given output pixel
__device__ inline float compute_conv_transpose_at_pixel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    int b, int in_channels, int in_height, int in_width,
    int out_channels, int kernel_size,
    int h_out, int w_out, int stride, int padding, int dilation, int o) {
  float sum = 0.0f;
  for (int c = 0; c < in_channels; ++c) {
    for (int p = 0; p < kernel_size; ++p) {
      int h_unscaled = h_out + padding - p * dilation;
      if (h_unscaled % stride != 0)
        continue;
      int h_in = h_unscaled / stride;
      if (h_in < 0 || h_in >= in_height)
        continue;
      for (int q = 0; q < kernel_size; ++q) {
        int w_unscaled = w_out + padding - q * dilation;
        if (w_unscaled % stride != 0)
          continue;
        int w_in = w_unscaled / stride;
        if (w_in < 0 || w_in >= in_width)
          continue;
        int input_idx = ((b * in_channels + c) * in_height + h_in) * in_width + w_in;
        int weight_idx = ((c * out_channels + o) * kernel_size + p) * kernel_size + q;
        sum += input[input_idx] * weight[weight_idx];
      }
    }
  }
  return sum;
}

// CUDA kernel for 2D transposed convolution using modular device functions
__global__ void conv_transpose2d_forward_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int batch_size,
    int in_channels,
    int out_channels,
    int in_height,
    int in_width,
    int kernel_size,
    int out_height,
    int out_width,
    int stride,
    int padding,
    int dilation) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total = batch_size * out_channels * out_height * out_width;
  if (index >= total)
    return;

  int b, o, h_out, w_out;
  decode_index(index, out_width, out_height, out_channels, b, o, h_out, w_out);

  // Initialize with bias
  float out_val = bias[o];
  // Add convolution contributions computed in a modular manner
  out_val += compute_conv_transpose_at_pixel(input, weight, b, in_channels, in_height, in_width,
                                              out_channels, kernel_size, h_out, w_out,
                                              stride, padding, dilation, o);

  int output_idx = ((b * out_channels + o) * out_height + h_out) * out_width + w_out;
  output[output_idx] = out_val;
}

// CUDA launcher function
torch::Tensor conv_transpose2d_forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    int stride,
    int padding,
    int dilation) {
  // Get input dimensions
  int batch_size = input.size(0);
  int in_channels = input.size(1);
  int in_height = input.size(2);
  int in_width = input.size(3);

  // Weight tensor: [in_channels, out_channels, kernel_size, kernel_size]
  int out_channels = weight.size(1);
  int kernel_size = weight.size(2); // assume square kernel

  // Calculate output dimensions
  int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;
  int out_width  = (in_width - 1) * stride - 2 * padding + dilation * (kernel_size - 1) + 1;

  auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, input.options());

  int total_threads = batch_size * out_channels * out_height * out_width;
  int threads = 1024;
  int blocks = (total_threads + threads - 1) / threads;

  conv_transpose2d_forward_kernel<<<blocks, threads>>>(
      input.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      output.data_ptr<float>(),
      batch_size,
      in_channels,
      out_channels,
      in_height,
      in_width,
      kernel_size,
      out_height,
      out_width,
      stride,
      padding,
      dilation);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in conv_transpose2d_forward_kernel: %s\n", hipGetErrorString(err));
  }

  return output;
}

// Wrapper to handle optional bias
torch::Tensor conv_transpose2d_forward_wrapper(
    torch::Tensor input,
    torch::Tensor weight,
    pybind11::object bias_obj,  // accepts None
    int stride,
    int padding,
    int dilation) {

  int out_channels = weight.size(1);
  torch::Tensor bias;
  if (bias_obj.is(pybind11::none())) {
    bias = torch::zeros({out_channels}, weight.options());
  } else {
    bias = bias_obj.cast<torch::Tensor>();
  }

  return conv_transpose2d_forward_cuda(input, weight, bias, stride, padding, dilation);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward", &conv_transpose2d_forward_wrapper,
        "ConvTranspose2d forward (CUDA) with modular device functions",
        pybind11::arg("input"),
        pybind11::arg("weight"),
        pybind11::arg("bias"),
        pybind11::arg("stride"),
        pybind11::arg("padding"),
        pybind11::arg("dilation"));
}
