#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

namespace py = pybind11;

// Utility function to parse int or sequence of ints from a pybind11 object
inline std::vector<int64_t> parseIntArrayRef(const py::object &obj) {
    std::vector<int64_t> result;
    if (py::isinstance<py::int_>(obj)) {
        result.push_back(obj.cast<int64_t>());
    } else if (py::isinstance<py::sequence>(obj)) {
        for (auto item : obj.cast<py::sequence>()) {
            result.push_back(py::cast<int64_t>(item));
        }
    } else {
        throw std::runtime_error("Expected int or sequence of ints");
    }
    return result;
}

// CUDA kernel using grid-stride loops to handle workloads larger than available threads
__global__ void conv_transpose2d_stride_loop_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,  // may be nullptr
    float* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int h_in,
    const int w_in,
    const int out_channels,
    const int h_out,
    const int w_out,
    const int kernel_size,
    const int stride_h,
    const int stride_w,
    const int padding_h,
    const int padding_w,
    const int groups,
    const int in_channels_per_group,
    const int out_channels_per_group,
    const int total_elements
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = blockDim.x * gridDim.x;
    for (int index = tid; index < total_elements; index += gridStride) {
        // Compute output indices (n, c, h, w) from linear index
        int w = index % w_out;
        int tmp = index / w_out;
        int h = tmp % h_out;
        tmp = tmp / h_out;
        int c = tmp % out_channels;
        int n = tmp / out_channels;

        int g = c / out_channels_per_group;
        int c_local = c % out_channels_per_group;

        float sum = 0.0f;
        // Loop over kernel spatial dimensions with manual unrolling
        #pragma unroll
        for (int kh = 0; kh < kernel_size; ++kh) {
            #pragma unroll
            for (int kw = 0; kw < kernel_size; ++kw) {
                int h_in_candidate = h + padding_h - kh;
                int w_in_candidate = w + padding_w - kw;
                // Check if the candidate position corresponds to a valid input index
                if ((h_in_candidate % stride_h == 0) && (w_in_candidate % stride_w == 0)) {
                    int h_in_idx = h_in_candidate / stride_h;
                    int w_in_idx = w_in_candidate / stride_w;
                    if (h_in_idx >= 0 && h_in_idx < h_in && w_in_idx >= 0 && w_in_idx < w_in) {
                        for (int r = 0; r < in_channels_per_group; ++r) {
                            int in_channel = g * in_channels_per_group + r;
                            int input_idx = ((n * in_channels + in_channel) * h_in + h_in_idx) * w_in + w_in_idx;
                            int weight_idx = (((g * in_channels_per_group + r) * out_channels_per_group + c_local) * kernel_size + kh) * kernel_size + kw;
                            float in_val = __ldg(&input[input_idx]);
                            float w_val  = __ldg(&weight[weight_idx]);
                            sum += in_val * w_val;
                        }
                    }
                }
            }
        }
        if (bias) {
            sum += __ldg(&bias[c]);
        }
        int output_idx = ((n * out_channels + c) * h_out + h) * w_out + w;
        output[output_idx] = sum;
    }
}

// Forward function callable from PyTorch
torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    c10::optional<torch::Tensor> bias,
    py::object stride = py::int_(1),
    py::object padding = py::int_(0),
    py::object output_padding = py::int_(0),
    int64_t groups = 1
) {
    auto stride_vec = parseIntArrayRef(stride);
    auto padding_vec = parseIntArrayRef(padding);
    auto output_padding_vec = parseIntArrayRef(output_padding);

    int stride_h = stride_vec[0];
    int stride_w = (stride_vec.size() > 1) ? stride_vec[1] : stride_h;
    int padding_h = padding_vec[0];
    int padding_w = (padding_vec.size() > 1) ? padding_vec[1] : padding_h;
    int output_padding_h = output_padding_vec[0];
    int output_padding_w = (output_padding_vec.size() > 1) ? output_padding_vec[1] : output_padding_h;

    // Input dimensions: [batch_size, in_channels, h_in, w_in]
    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int h_in = x.size(2);
    const int w_in = x.size(3);

    // Weight dimensions: [in_channels, out_channels_per_group, kernel_size, kernel_size]
    const int kernel_size = weight.size(2); // assuming square kernel
    int out_channels = weight.size(1) * groups;

    // Compute output dimensions for transposed convolution
    int h_out = (h_in - 1) * stride_h - 2 * padding_h + kernel_size + output_padding_h;
    int w_out = (w_in - 1) * stride_w - 2 * padding_w + kernel_size + output_padding_w;

    auto output_tensor = torch::zeros({batch_size, out_channels, h_out, w_out}, x.options());
    int in_channels_per_group = in_channels / groups;
    int out_channels_per_group = out_channels / groups;

    int total_elements = batch_size * out_channels * h_out * w_out;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;

    const float* input_ptr = x.data_ptr<float>();
    const float* weight_ptr = weight.data_ptr<float>();
    const float* bias_ptr = bias.has_value() ? bias.value().data_ptr<float>() : nullptr;
    float* output_ptr = output_tensor.data_ptr<float>();

    conv_transpose2d_stride_loop_kernel<<<grid_size, block_size>>>(
        input_ptr,
        weight_ptr,
        bias_ptr,
        output_ptr,
        batch_size,
        in_channels,
        h_in,
        w_in,
        out_channels,
        h_out,
        w_out,
        kernel_size,
        stride_h,
        stride_w,
        padding_h,
        padding_w,
        groups,
        in_channels_per_group,
        out_channels_per_group,
        total_elements
    );

    hipDeviceSynchronize();
    return output_tensor;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "ConvTranspose2d forward with grid-stride loops for large workloads",
          py::arg("x"),
          py::arg("weight"),
          py::arg("bias") = py::none(),
          py::arg("stride") = 1,
          py::arg("padding") = 0,
          py::arg("output_padding") = 0,
          py::arg("groups") = 1);
}
