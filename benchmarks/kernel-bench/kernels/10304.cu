#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void forward_kernel_atomic_minimization(
    const float* x,
    const float* weight,
    const float* bias,
    float* output,
    int B,
    int IC,
    int OC,
    int H,
    int W
) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_elements = B * OC * H * W;
    
    if (index >= total_elements) return;
    
    // Decompose linear index into 4D tensor coordinates
    int w = index % W;
    int h = (index / W) % H;
    int oc = (index / (W * H)) % OC;
    int b = index / (W * H * OC);

    float sum = 0.0f;
    
    // Allocate shared memory to minimize global memory atomic operations
    extern __shared__ float shared_sum[];
    shared_sum[threadIdx.x] = 0.0f;
    __syncthreads();
    
    // 1x1 convolution equivalent to matmul over channels
    for (int ic = 0; ic < IC; ++ic) {
        const int x_offset = b * IC * H * W + ic * H * W + h * W + w;
        const int w_offset = oc * IC + ic;
        atomicAdd(&shared_sum[threadIdx.x], x[x_offset] * weight[w_offset]);
    }

    __syncthreads();

    // Reduce shared memory to global memory
    if (threadIdx.x == 0) {
        for (int i = 0; i < blockDim.x; ++i) {
            sum += shared_sum[i];
        }
        // Handle optional bias
        output[index] = sum + (bias ? bias[oc] : 0.0f);
    }
}

torch::Tensor forward_cuda_atomic_minimization(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias
) {
    // Input validation
    TORCH_CHECK(x.is_cuda() && weight.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(x.dim() == 4, "x must be 4D (NCHW)");
    TORCH_CHECK(weight.dim() == 4, "Weight must be 4D (OC, IC, 1, 1)");
    if (bias) {
        TORCH_CHECK(bias->is_cuda(), "Bias must be CUDA tensor");
        TORCH_CHECK(bias->dim() == 1, "Bias must be 1D");
    }

    const int B = x.size(0);
    const int IC = x.size(1);
    const int H = x.size(2);
    const int W = x.size(3);
    const int OC = weight.size(0);

    TORCH_CHECK(weight.size(1) == IC, "Input/output channel mismatch");
    TORCH_CHECK(weight.size(2) == 1 && weight.size(3) == 1, "Kernel must be 1x1");
    if (bias) {
        TORCH_CHECK(bias->size(0) == OC, "Bias/out channel mismatch");
    }

    // Create output tensor
    auto output = torch::empty({B, OC, H, W}, x.options());

    // Get raw pointers
    const float* x_ptr = x.data_ptr<float>();
    const float* w_ptr = weight.data_ptr<float>();
    const float* b_ptr = bias ? bias->data_ptr<float>() : nullptr;
    float* out_ptr = output.data_ptr<float>();

    // Launch kernel
    const int threads = 256;
    const int blocks = (B * OC * H * W + threads - 1) / threads;
    const int shared_mem_size = threads * sizeof(float);
    
    forward_kernel_atomic_minimization<<<blocks, threads, shared_mem_size>>>(
        x_ptr, w_ptr, b_ptr, out_ptr,
        B, IC, OC, H, W
    );
    
    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_cuda_atomic_minimization, "Pointwise 2D convolution forward with minimized atomic operations (CUDA)");
}