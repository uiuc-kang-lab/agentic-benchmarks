#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__inline__ __device__ float warpReduceSum(float val) {
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float sum = 0.0f;

    if (row < N && col < N) {
        if (row < col) {
            C[row * N + col] = 0.f;
        } else {
            for (int t = col/TILE_SIZE; t <= row/TILE_SIZE; t++) {
                float a_val = 0.0f;
                float b_val = 0.0f;

                if (row < N && (t*TILE_SIZE + tx) <= row) {
                    a_val = A[row * N + (t*TILE_SIZE + tx)];
                }
                if ((t*TILE_SIZE + ty) < N && col < N) {
                    b_val = B[(t*TILE_SIZE + ty) * N + col];
                }

                sum += a_val * b_val;
            }

            // Use warp-level reduction to sum across threads in a warp
            sum = warpReduceSum(sum);

            // Use first thread in each warp to write the result
            if (tx % warpSize == 0) {
                C[row * N + col] = sum;
            }
        }
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE, 
                   (N + TILE_SIZE - 1) / TILE_SIZE);

    hipStream_t stream;
    hipStreamCreate(&stream);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Warp-level optimized triangular matrix multiplication (CUDA)");
}