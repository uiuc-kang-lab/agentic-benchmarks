#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel using a grid-stride loop to reduce overhead
// and avoid unnecessary synchronizations since each thread operates independently
template <typename scalar_t>
__global__ void relu_kernel_optimized(
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ input,
    const int64_t size) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Each thread processes multiple elements in a grid-stride loop
    for (; idx < size; idx += stride) {
        scalar_t x = input[idx];
        output[idx] = (x > 0) ? x : static_cast<scalar_t>(0);
    }
}

// PyTorch wrapper function
torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    
    const int threads = 256;
    const int blocks = (input.numel() + threads - 1) / threads;
    
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "relu_kernel_optimized", ([&] {
        relu_kernel_optimized<scalar_t><<<blocks, threads>>>(
            output.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            input.numel()
        );
    }));
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized ReLU forward (CUDA)");
}
