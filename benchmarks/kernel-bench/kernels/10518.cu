#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void cumsum_kernel(const float* input, float* output, int total_pairs, int inner_size, int stride) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= total_pairs) return;
    
    int outer_idx = tid / inner_size;
    int inner_idx = tid % inner_size;
    
    float sum = 0.0f;
    for (int i = 0; i < stride; ++i) {
        int idx = outer_idx * stride * inner_size + i * inner_size + inner_idx;
        sum += input[idx];
        output[idx] = sum;
    }
}

torch::Tensor forward(torch::Tensor x, int dim) {
    CHECK_INPUT(x);

    auto output = torch::empty_like(x);
    int ndim = x.dim();
    dim = (dim + ndim) % ndim;

    int outer_size = 1;
    for (int i = 0; i < dim; ++i) {
        outer_size *= x.size(i);
    }

    int inner_size = 1;
    for (int i = dim + 1; i < ndim; ++i) {
        inner_size *= x.size(i);
    }

    int stride = x.size(dim);
    int total_pairs = outer_size * inner_size;
    int threads_per_block = 256;
    int blocks = (total_pairs + threads_per_block - 1) / threads_per_block;

    cumsum_kernel<<<blocks, threads_per_block>>>(
        x.data_ptr<float>(), output.data_ptr<float>(), total_pairs, inner_size, stride
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "CUDA cumulative sum optimized");
}