#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32
#define THREAD_TILE 2
#define MAX_MATRIX_DIM 8192

// Constant memory for matrix dimensions and tile count
__constant__ int d_N;
__constant__ int d_num_tiles;

// Optimized kernel using __ldg() for read-only global memory accesses and aligning accesses to 128-bit boundaries
__global__ void matmul_kernel_aligned(const float* __restrict__ A,
                                      const float* __restrict__ B,
                                      float* __restrict__ C) {
    // Determine the starting row and column for this block
    int blockRow = blockIdx.y * BLOCK_SIZE;
    int blockCol = blockIdx.x * BLOCK_SIZE;

    // Each thread computes a 2x2 sub-tile
    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int row = blockRow + ty * THREAD_TILE;
    int col = blockCol + tx * THREAD_TILE;

    // Registers for the 2x2 sub-tile results
    float regC00 = 0.0f, regC01 = 0.0f, regC10 = 0.0f, regC11 = 0.0f;

    // Shared memory tiles for A and B
    __shared__ float s_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float s_B[BLOCK_SIZE][BLOCK_SIZE];

    // Loop over all tiles in the k-dimension
    for (int t = 0; t < d_num_tiles; t++) {
        // Calculate indices for loading a 2x2 block from A
        int aRow0 = row;
        int aRow1 = row + 1;
        int aCol0 = t * BLOCK_SIZE + tx * THREAD_TILE;
        int aCol1 = aCol0 + 1;

        // Load elements from A using __ldg() for read-only caching
        s_A[ty * THREAD_TILE + 0][tx * THREAD_TILE + 0] = (aRow0 < d_N && aCol0 < d_N) ? __ldg(&A[aRow0 * d_N + aCol0]) : 0.0f;
        s_A[ty * THREAD_TILE + 0][tx * THREAD_TILE + 1] = (aRow0 < d_N && aCol1 < d_N) ? __ldg(&A[aRow0 * d_N + aCol1]) : 0.0f;
        s_A[ty * THREAD_TILE + 1][tx * THREAD_TILE + 0] = (aRow1 < d_N && aCol0 < d_N) ? __ldg(&A[aRow1 * d_N + aCol0]) : 0.0f;
        s_A[ty * THREAD_TILE + 1][tx * THREAD_TILE + 1] = (aRow1 < d_N && aCol1 < d_N) ? __ldg(&A[aRow1 * d_N + aCol1]) : 0.0f;

        // Calculate indices for loading a 2x2 block from B
        int bRow0 = t * BLOCK_SIZE + ty * THREAD_TILE;
        int bRow1 = bRow0 + 1;
        int bCol0 = col;
        int bCol1 = col + 1;

        // Load elements from B using __ldg()
        s_B[ty * THREAD_TILE + 0][tx * THREAD_TILE + 0] = (bRow0 < d_N && bCol0 < d_N) ? __ldg(&B[bRow0 * d_N + bCol0]) : 0.0f;
        s_B[ty * THREAD_TILE + 0][tx * THREAD_TILE + 1] = (bRow0 < d_N && bCol1 < d_N) ? __ldg(&B[bRow0 * d_N + bCol1]) : 0.0f;
        s_B[ty * THREAD_TILE + 1][tx * THREAD_TILE + 0] = (bRow1 < d_N && bCol0 < d_N) ? __ldg(&B[bRow1 * d_N + bCol0]) : 0.0f;
        s_B[ty * THREAD_TILE + 1][tx * THREAD_TILE + 1] = (bRow1 < d_N && bCol1 < d_N) ? __ldg(&B[bRow1 * d_N + bCol1]) : 0.0f;

        __syncthreads();

        // Multiply the loaded tile and accumulate results for the 2x2 sub-tile
        for (int k = 0; k < BLOCK_SIZE; k++) {
            float a0 = s_A[ty * THREAD_TILE + 0][k];
            float a1 = s_A[ty * THREAD_TILE + 1][k];
            float b0 = s_B[k][tx * THREAD_TILE + 0];
            float b1 = s_B[k][tx * THREAD_TILE + 1];
            regC00 += a0 * b0;
            regC01 += a0 * b1;
            regC10 += a1 * b0;
            regC11 += a1 * b1;
        }

        __syncthreads();
    }

    // Write the 2x2 block results back to global memory with boundary checking
    if (row < d_N && col < d_N)
        C[row * d_N + col] = regC00;
    if (row < d_N && (col + 1) < d_N)
        C[row * d_N + col + 1] = regC01;
    if ((row + 1) < d_N && col < d_N)
        C[(row + 1) * d_N + col] = regC10;
    if ((row + 1) < d_N && (col + 1) < d_N)
        C[(row + 1) * d_N + col + 1] = regC11;
}

// C++ interface (Pybind11 binding)
torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D matrices");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must have the same dimensions");
    TORCH_CHECK(A.size(0) <= MAX_MATRIX_DIM, "Matrix dimension exceeds maximum supported size");

    int N = A.size(0);
    int num_tiles = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipMemcpyToSymbol(HIP_SYMBOL(d_N), &N, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_tiles), &num_tiles, sizeof(int));

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, A.get_device());
    auto C = torch::zeros({N, N}, options);

    // Each block uses (BLOCK_SIZE/THREAD_TILE) x (BLOCK_SIZE/THREAD_TILE) threads
    dim3 threads(BLOCK_SIZE / THREAD_TILE, BLOCK_SIZE / THREAD_TILE);
    dim3 blocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matmul_kernel_aligned<<<blocks, threads>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>());

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Vectorized Read-Only Matrix Multiplication with __ldg (CUDA)");
}
