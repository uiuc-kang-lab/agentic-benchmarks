#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// This kernel uses __ldg() for read-only global memory loads to improve cache utilization
// and ensures that accesses are aligned to 128-bit boundaries when possible.

template <typename scalar_t>
__global__ void conv_transpose2d_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int in_channels,
    const int in_height,
    const int in_width,
    const int out_channels,
    const int kernel_h,
    const int kernel_w,
    const int stride,
    const int padding,
    const int output_padding,
    const int groups,
    const int dilation,
    const int out_height,
    const int out_width
) {
    const int total_elements = batch_size * out_channels * out_height * out_width;
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_elements) return;

    // Unravel the flat index into 4D indices
    int n = idx;
    const int ow = n % out_width;
    n /= out_width;
    const int oh = n % out_height;
    n /= out_height;
    const int oc = n % out_channels;
    n /= out_channels;
    const int b = n;

    if (b >= batch_size) return;

    // Group and channel calculations
    const int out_channels_per_group = out_channels / groups;
    const int g = oc / out_channels_per_group;
    const int oc_group = oc % out_channels_per_group;  
    const int in_channels_per_group = in_channels / groups;
    const int ic_start = g * in_channels_per_group;

    // Use __ldg() for read-only bias load (if provided)
    scalar_t val = (bias != nullptr) ? __ldg(bias + oc) : static_cast<scalar_t>(0);

    // Pre-calculate common strides for aligned memory accesses
    const int input_channel_stride = in_height * in_width;
    const int weight_channel_stride = kernel_h * kernel_w;

    // Loop over the kernel window
    for (int kh = 0; kh < kernel_h; ++kh) {
        int h_in_base = oh - kh * dilation + padding;
        if (h_in_base % stride != 0) continue;
        int h_in = h_in_base / stride;
        if (h_in < 0 || h_in >= in_height) continue;
        
        for (int kw = 0; kw < kernel_w; ++kw) {
            int w_in_base = ow - kw * dilation + padding;
            if (w_in_base % stride != 0) continue;
            int w_in = w_in_base / stride;
            if (w_in < 0 || w_in >= in_width) continue;
            
            // Loop over the channels in the current group
            for (int ic = 0; ic < in_channels_per_group; ++ic) {
                int input_idx = b * in_channels * in_height * in_width +
                                (ic_start + ic) * input_channel_stride +
                                h_in * in_width + w_in;
                int weight_idx = (ic_start + ic) * (out_channels_per_group * kernel_h * kernel_w) +
                                 oc_group * weight_channel_stride +
                                 kh * kernel_w + kw;
                
                // Use __ldg() for read-only loads of input and weight
                scalar_t x_val = __ldg(input + input_idx);
                scalar_t w_val = __ldg(weight + weight_idx);
                val += x_val * w_val;
            }
        }
    }
    
    output[idx] = val;
}


torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias,
    int stride,
    int padding,
    int output_padding,
    int groups,
    int dilation = 1
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(x.dim() == 4, "Input must be 4D");
    TORCH_CHECK(weight.dim() == 4, "Weight must be 4D");

    const int batch_size = x.size(0);
    const int in_channels = x.size(1);
    const int in_height = x.size(2);
    const int in_width = x.size(3);

    const int out_channels = weight.size(1) * groups;
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);

    if (bias.has_value() && bias->defined()) {
        TORCH_CHECK(bias->numel() == out_channels, "Bias must have out_channels elements");
        TORCH_CHECK(bias->device().is_cuda(), "Bias must be a CUDA tensor");
    }

    const int out_height = (in_height - 1) * stride - 2 * padding + dilation * (kernel_h - 1) + output_padding + 1;
    const int out_width = (in_width - 1) * stride - 2 * padding + dilation * (kernel_w - 1) + output_padding + 1;

    auto output = torch::zeros({batch_size, out_channels, out_height, out_width}, x.options());

    const int total_elements = output.numel();
    const int threads = 256;
    const int blocks = (total_elements + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "conv_transpose2d_cuda", ([&] {
        conv_transpose2d_kernel<scalar_t><<<blocks, threads>>>(
            x.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            (bias.has_value() && bias->defined()) ? bias->data_ptr<scalar_t>() : nullptr,
            output.data_ptr<scalar_t>(),
            batch_size,
            in_channels,
            in_height,
            in_width,
            out_channels,
            kernel_h,
            kernel_w,
            stride,
            padding,
            output_padding,
            groups,
            dilation,
            out_height,
            out_width
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Transposed 2D convolution optimized with __ldg (CUDA)",
          py::arg("x"), py::arg("weight"), py::arg("bias") = py::none(),
          py::arg("stride"), py::arg("padding"), py::arg("output_padding"),
          py::arg("groups"), py::arg("dilation") = 1);
}
