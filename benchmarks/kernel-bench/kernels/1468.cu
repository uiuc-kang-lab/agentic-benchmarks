#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32
#define THREAD_TILE 4
#define MAX_MATRIX_DIM 8192

// Constant memory for matrix dimensions and number of tiles
__constant__ int d_N;
__constant__ int d_num_tiles;

// This kernel employs grid-stride loops to distribute workload evenly across threads and blocks.
// It uses vectorized 128-bit aligned loads with __ldg() to load data efficiently via float4.
// Each thread computes a 4x4 sub-tile of the output. The grid-stride loops allow each block to process
// multiple output tiles if necessary, reducing potential bottlenecks due to uneven workload distribution.

__global__ void grid_stride_vec_ldg_matmul(const float* __restrict__ A,
                                            const float* __restrict__ B,
                                            float* __restrict__ C) {
    // Shared memory tiles for A and B
    __shared__ float s_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float s_B[BLOCK_SIZE][BLOCK_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Number of tiles per dimension
    int numTiles = d_num_tiles;  // (N + BLOCK_SIZE - 1) / BLOCK_SIZE

    // Grid-stride loops over the output tile blocks
    for (int tileRow = blockIdx.y; tileRow < numTiles; tileRow += gridDim.y) {
        for (int tileCol = blockIdx.x; tileCol < numTiles; tileCol += gridDim.x) {
            // Each thread computes a 4x4 sub-tile of the output tile
            float regC[THREAD_TILE][THREAD_TILE] = { {0.f, 0.f, 0.f, 0.f},
                                                       {0.f, 0.f, 0.f, 0.f},
                                                       {0.f, 0.f, 0.f, 0.f},
                                                       {0.f, 0.f, 0.f, 0.f} };
            
            // Loop over k-dimension tiles
            for (int t = 0; t < numTiles; t++) {
                // Load A tile: from global A tile at (tileRow * BLOCK_SIZE, t * BLOCK_SIZE) into s_A
                int total_A_loads = (BLOCK_SIZE * BLOCK_SIZE) / 4;  // each load uses float4
                int tid = ty * blockDim.x + tx;
                for (int i = tid; i < total_A_loads; i += (blockDim.x * blockDim.y)) {
                    int a_row_in_tile = i / (BLOCK_SIZE / 4); // BLOCK_SIZE/4 = 8
                    int a_col_group = i % (BLOCK_SIZE / 4);
                    int a_global_row = tileRow * BLOCK_SIZE + a_row_in_tile;
                    int a_global_col = t * BLOCK_SIZE + a_col_group * 4;
                    float4 A_vec;
                    if (a_global_row < d_N && (a_global_col + 3) < d_N) {
                        const float4* A_vec_ptr = reinterpret_cast<const float4*>(A);
                        int index = a_global_row * d_N + a_global_col;
                        A_vec = __ldg(&A_vec_ptr[index / 4]);
                    } else {
                        float tmp[4] = {0.f, 0.f, 0.f, 0.f};
                        for (int j = 0; j < 4; j++) {
                            int col = a_global_col + j;
                            if (a_global_row < d_N && col < d_N)
                                tmp[j] = __ldg(&A[a_global_row * d_N + col]);
                        }
                        A_vec.x = tmp[0]; A_vec.y = tmp[1]; A_vec.z = tmp[2]; A_vec.w = tmp[3];
                    }
                    int dest_col = a_col_group * 4;
                    s_A[a_row_in_tile][dest_col + 0] = A_vec.x;
                    s_A[a_row_in_tile][dest_col + 1] = A_vec.y;
                    s_A[a_row_in_tile][dest_col + 2] = A_vec.z;
                    s_A[a_row_in_tile][dest_col + 3] = A_vec.w;
                }

                // Load B tile: from global B tile at (t * BLOCK_SIZE, tileCol * BLOCK_SIZE) into s_B
                int total_B_loads = (BLOCK_SIZE * BLOCK_SIZE) / 4;
                for (int i = tid; i < total_B_loads; i += (blockDim.x * blockDim.y)) {
                    int b_row_in_tile = i / (BLOCK_SIZE / 4);
                    int b_col_group = i % (BLOCK_SIZE / 4);
                    int b_global_row = t * BLOCK_SIZE + b_row_in_tile;
                    int b_global_col = tileCol * BLOCK_SIZE + b_col_group * 4;
                    float4 B_vec;
                    if (b_global_row < d_N && (b_global_col + 3) < d_N) {
                        const float4* B_vec_ptr = reinterpret_cast<const float4*>(B);
                        int index = b_global_row * d_N + b_global_col;
                        B_vec = __ldg(&B_vec_ptr[index / 4]);
                    } else {
                        float tmp[4] = {0.f, 0.f, 0.f, 0.f};
                        for (int j = 0; j < 4; j++) {
                            int col = b_global_col + j;
                            if (b_global_row < d_N && col < d_N)
                                tmp[j] = __ldg(&B[b_global_row * d_N + col]);
                        }
                        B_vec.x = tmp[0]; B_vec.y = tmp[1]; B_vec.z = tmp[2]; B_vec.w = tmp[3];
                    }
                    int dest_col = b_col_group * 4;
                    s_B[b_row_in_tile][dest_col + 0] = B_vec.x;
                    s_B[b_row_in_tile][dest_col + 1] = B_vec.y;
                    s_B[b_row_in_tile][dest_col + 2] = B_vec.z;
                    s_B[b_row_in_tile][dest_col + 3] = B_vec.w;
                }
                __syncthreads();

                // Compute the multiplication for the current tile
                int a_sub_row = ty * THREAD_TILE;
                int b_sub_col = tx * THREAD_TILE;
                #pragma unroll
                for (int k = 0; k < BLOCK_SIZE; k++) {
                    float a0 = s_A[a_sub_row + 0][k];
                    float a1 = s_A[a_sub_row + 1][k];
                    float a2 = s_A[a_sub_row + 2][k];
                    float a3 = s_A[a_sub_row + 3][k];

                    float b0 = s_B[k][b_sub_col + 0];
                    float b1 = s_B[k][b_sub_col + 1];
                    float b2 = s_B[k][b_sub_col + 2];
                    float b3 = s_B[k][b_sub_col + 3];

                    regC[0][0] += a0 * b0;
                    regC[0][1] += a0 * b1;
                    regC[0][2] += a0 * b2;
                    regC[0][3] += a0 * b3;

                    regC[1][0] += a1 * b0;
                    regC[1][1] += a1 * b1;
                    regC[1][2] += a1 * b2;
                    regC[1][3] += a1 * b3;

                    regC[2][0] += a2 * b0;
                    regC[2][1] += a2 * b1;
                    regC[2][2] += a2 * b2;
                    regC[2][3] += a2 * b3;

                    regC[3][0] += a3 * b0;
                    regC[3][1] += a3 * b1;
                    regC[3][2] += a3 * b2;
                    regC[3][3] += a3 * b3;
                }
                __syncthreads();
            } // end loop over t
            
            // Write the computed 4x4 sub-tile to global memory
            for (int i = 0; i < THREAD_TILE; i++) {
                int global_row = tileRow * BLOCK_SIZE + ty * THREAD_TILE + i;
                if (global_row < d_N) {
                    int global_col = tileCol * BLOCK_SIZE + tx * THREAD_TILE;
                    if (global_col + 3 < d_N) {
                        float4 out_val;
                        out_val.x = regC[i][0];
                        out_val.y = regC[i][1];
                        out_val.z = regC[i][2];
                        out_val.w = regC[i][3];
                        float4* C_vec_ptr = reinterpret_cast<float4*>(C);
                        int index = global_row * d_N + global_col;
                        C_vec_ptr[index / 4] = out_val;
                    } else {
                        for (int j = 0; j < THREAD_TILE; j++) {
                            int global_col_j = global_col + j;
                            if (global_col_j < d_N)
                                C[global_row * d_N + global_col_j] = regC[i][j];
                        }
                    }
                }
            }
            __syncthreads();
        } // end for tileCol
    } // end for tileRow
}

// C++ interface (Pybind11 binding)

torch::Tensor forward(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "A and B must be 2D matrices");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must have the same dimensions");
    TORCH_CHECK(A.size(0) <= MAX_MATRIX_DIM, "Matrix dimension exceeds maximum supported size");

    int N = A.size(0);
    int num_tiles = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    hipMemcpyToSymbol(HIP_SYMBOL(d_N), &N, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_tiles), &num_tiles, sizeof(int));

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, A.get_device());
    auto C = torch::zeros({N, N}, options);

    // Launch configuration:
    // Each block has (BLOCK_SIZE/THREAD_TILE, BLOCK_SIZE/THREAD_TILE) threads, i.e., (8,8)
    // Grid dimensions set to cover all output tiles
    dim3 threads(BLOCK_SIZE / THREAD_TILE, BLOCK_SIZE / THREAD_TILE);
    dim3 blocks(num_tiles, num_tiles);
    
    grid_stride_vec_ldg_matmul<<<blocks, threads>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>());
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Grid-Stride Vectorized 128-bit Aligned Matrix Multiplication (CUDA)");
}
