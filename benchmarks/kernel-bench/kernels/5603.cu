#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <limits>

template <typename scalar_t, int KERNEL_SIZE>
__global__ void max_pool2d_optimized_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const int stride,
    const int padding,
    const int dilation
) {
    // Spatial dimensions mapped to 2D block for coalescing
    int ow = blockIdx.x * blockDim.x + threadIdx.x;
    int oh = blockIdx.y * blockDim.y + threadIdx.y;
    int bc = blockIdx.z;
    if (ow >= output_width || oh >= output_height) return;

    // Decompose batch-channel index
    int b = bc / channels;
    int c = bc % channels;
    
    // Precompute input offsets
    const int input_base = b * channels * input_height * input_width
                        + c * input_height * input_width;
    const int ih_start = oh * stride - padding;
    const int iw_start = ow * stride - padding;

    // Optimized loop with boundary checks
    scalar_t max_val = -std::numeric_limits<scalar_t>::infinity();
    if constexpr (KERNEL_SIZE == 2) {
        #pragma unroll
        for (int kh = 0; kh < 2; ++kh) {
            int ih = ih_start + kh * dilation;
            if (ih >= 0 && ih < input_height) {
                int row_off = ih * input_width;
                #pragma unroll
                for (int kw = 0; kw < 2; ++kw) {
                    int iw = iw_start + kw * dilation;
                    if (iw >= 0 && iw < input_width)
                        max_val = fmaxf(max_val, __ldg(input + input_base + row_off + iw));
                }
            }
        }
    }
    else if constexpr (KERNEL_SIZE == 3) {
        #pragma unroll
        for (int kh = 0; kh < 3; ++kh) {
            int ih = ih_start + kh * dilation;
            if (ih >= 0 && ih < input_height) {
                int row_off = ih * input_width;
                #pragma unroll
                for (int kw = 0; kw < 3; ++kw) {
                    int iw = iw_start + kw * dilation;
                    if (iw >= 0 && iw < input_width)
                        max_val = fmaxf(max_val, __ldg(input + input_base + row_off + iw));
                }
            }
        }
    }
    else {
        for (int kh = 0; kh < KERNEL_SIZE; ++kh) {
            int ih = ih_start + kh * dilation;
            if (ih >= 0 && ih < input_height) {
                int row_off = ih * input_width;
                for (int kw = 0; kw < KERNEL_SIZE; ++kw) {
                    int iw = iw_start + kw * dilation;
                    if (iw >= 0 && iw < input_width)
                        max_val = fmaxf(max_val, __ldg(input + input_base + row_off + iw));
                }
            }
        }
    }

    // Coalesced write with spatial locality
    output[b * channels * output_height * output_width
         + c * output_height * output_width
         + oh * output_width
         + ow] = max_val;
}

torch::Tensor max_pool2d_optimized_forward(
    torch::Tensor input,
    int kernel_size,
    int stride,
    int padding,
    int dilation
) {
    const auto batch_size = input.size(0);
    const auto channels = input.size(1);
    const auto input_height = input.size(2);
    const auto input_width = input.size(3);
    const auto output_height = (input_height + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;
    const auto output_width = (input_width + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;

    auto output = torch::empty({batch_size, channels, output_height, output_width}, input.options());

    dim3 threads(16, 16);  // Better memory coalescing
    dim3 blocks(
        (output_width + threads.x - 1) / threads.x,
        (output_height + threads.y - 1) / threads.y,
        batch_size * channels  // Natural bc grouping
    );

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "max_pool_forward", ([&] {
        if (kernel_size == 2) {
            max_pool2d_optimized_kernel<scalar_t, 2><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size, channels,
                input_height, input_width,
                output_height, output_width,
                stride, padding, dilation);
        } else if (kernel_size == 3) {
            max_pool2d_optimized_kernel<scalar_t, 3><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size, channels,
                input_height, input_width,
                output_height, output_width,
                stride, padding, dilation);
        } else {
            max_pool2d_optimized_kernel<scalar_t, -1><<<blocks, threads>>>(
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size, channels,
                input_height, input_width,
                output_height, output_width,
                stride, padding, dilation);
        }
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &max_pool2d_optimized_forward, "Max Pool 2D Optimized forward (CUDA)");
}