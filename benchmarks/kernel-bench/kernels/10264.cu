#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void forward_kernel(
    const float* __restrict__ x,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    int B,
    int IC,
    int OC,
    int H,
    int W
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int total_elements = B * OC * H * W;
    
    // Align IC to 4 elements (16 bytes) for better memory access
    const int IC_aligned = ((IC + 3) / 4) * 4;
    
    for (int index = tid; index < total_elements; index += stride) {
        const int w = index % W;
        const int h = (index / W) % H;
        const int oc = (index / (W * H)) % OC;
        const int b = index / (W * H * OC);

        float sum = 0.0f;
        
        // Process all input channels
        for (int ic = 0; ic < IC; ic++) {
            const int x_offset = b * IC * H * W + ic * H * W + h * W + w;
            const int w_offset = oc * IC + ic;
            sum += __ldg(&x[x_offset]) * __ldg(&weight[w_offset]);
        }
        
        // Handle bias with __ldg
        if (bias) {
            sum += __ldg(&bias[oc]);
        }
        
        output[index] = sum;
    }
}

torch::Tensor forward_cuda(
    torch::Tensor x,
    torch::Tensor weight,
    torch::optional<torch::Tensor> bias
) {
    TORCH_CHECK(x.is_cuda() && weight.is_cuda(), "Inputs must be CUDA tensors");
    TORCH_CHECK(x.dim() == 4, "x must be 4D (NCHW)");
    TORCH_CHECK(weight.dim() == 4, "Weight must be 4D (OC, IC, 1, 1)");
    
    const int B = x.size(0);
    const int IC = x.size(1);
    const int H = x.size(2);
    const int W = x.size(3);
    const int OC = weight.size(0);

    if (bias) {
        TORCH_CHECK(bias->is_cuda(), "Bias must be CUDA tensor");
        TORCH_CHECK(bias->dim() == 1, "Bias must be 1D");
        TORCH_CHECK(bias->size(0) == OC, "Bias/out channel mismatch");
    }

    TORCH_CHECK(weight.size(1) == IC, "Input/output channel mismatch");
    TORCH_CHECK(weight.size(2) == 1 && weight.size(3) == 1, "Kernel must be 1x1");

    auto output = torch::empty({B, OC, H, W}, x.options());

    const float* x_ptr = x.data_ptr<float>();
    const float* w_ptr = weight.data_ptr<float>();
    const float* b_ptr = bias ? bias->data_ptr<float>() : nullptr;
    float* out_ptr = output.data_ptr<float>();

    // Optimize grid size based on SM count
    int dev_id;
    hipGetDevice(&dev_id);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, dev_id);
    
    const int threads = 256;
    const int blocks = min(props.multiProcessorCount * 32,
                         (B * OC * H * W + threads - 1) / threads);
    
    forward_kernel<<<blocks, threads>>>(
        x_ptr, w_ptr, b_ptr, out_ptr,
        B, IC, OC, H, W
    );
    
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward_cuda, "Pointwise 2D convolution forward (CUDA)");
}