#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                   const float* __restrict__ B,
                                   float* __restrict__ C,
                                   const int N) {
    // Use 2D thread blocks for better memory access patterns
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row >= N || col >= N) return;

    // Pre-calculate the valid range for this row
    const int k_start = (row >= col) ? col : N;
    const int k_end = row;
    
    // Initialize result
    float sum = 0.0f;
    
    // Only compute if we're in the lower triangle and have valid k range
    if (k_start <= k_end) {
        // Compute matrix multiplication for this element
        #pragma unroll 4
        for (int k = k_start; k <= k_end; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
    }
    
    // Write result - no branch needed as sum will be 0 for upper triangle
    C[row * N + col] = sum;
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    // Use 16x16 thread blocks for good occupancy and memory coalescing
    const int BLOCK_SIZE = 16;
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, 
                   (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Triangular matrix multiplication (CUDA)");
}