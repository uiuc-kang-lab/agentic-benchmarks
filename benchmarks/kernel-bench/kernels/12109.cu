#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 128

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void hinge_loss_kernel(const float* __restrict__ predictions, const float* __restrict__ targets, float* __restrict__ output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (; idx < n; idx += stride) {
        output[idx] = fmaxf(0.0f, 1.0f - predictions[idx] * targets[idx]);
    }
}

torch::Tensor forward(torch::Tensor predictions, torch::Tensor targets) {
    CHECK_INPUT(predictions);
    CHECK_INPUT(targets);

    int n = predictions.numel();
    torch::Tensor output = torch::empty_like(predictions);

    int threads = BLOCK_SIZE;  // Experimenting with 128 threads per block
    int blocks = (n + threads - 1) / threads;

    hinge_loss_kernel<<<blocks, threads>>>(
        predictions.data_ptr<float>(),
        targets.data_ptr<float>(),
        output.data_ptr<float>(),
        n
    );

    // Compute the mean hinge loss
    auto mean = torch::mean(output);
    return mean;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Hinge Loss Forward with optimized block size (128 threads per block)");
}
