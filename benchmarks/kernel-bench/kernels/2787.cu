#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel that computes the Sigmoid activation, using a tuned block size based on performance experiments.
// We use a grid-stride loop to cover all elements in the tensor. The block size is set to 512.

template <typename scalar_t>
__global__ void sigmoid_kernel(const scalar_t* __restrict__ input,
                               scalar_t* __restrict__ output,
                               const int64_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = idx; i < size; i += stride) {
        // Compute sigmoid using full precision float computations
        float val = static_cast<float>(input[i]);
        float exp_val = expf(-val);
        float res = 1.0f / (1.0f + exp_val);
        output[i] = static_cast<scalar_t>(res);
    }
}

// Forward function to be called from Python
torch::Tensor forward(torch::Tensor input) {
    auto output = torch::empty_like(input);
    const int64_t size = input.numel();

    // Experimentally tuned block size based on NVIDIA H100 characteristics
    constexpr int THREADS = 512; 
    const int blocks = (size + THREADS - 1) / THREADS;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sigmoid_kernel", ([&] {
        const auto* input_data = input.data_ptr<scalar_t>();
        auto* output_data = output.data_ptr<scalar_t>();

        sigmoid_kernel<scalar_t><<<blocks, THREADS>>>(input_data, output_data, size);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Sigmoid forward (CUDA) tuned block size");
}
