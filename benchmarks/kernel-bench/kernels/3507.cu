#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Device helper: define an inline exponential function for float and double.
template <typename scalar_t>
__device__ inline scalar_t my_exp(scalar_t x);

template <>
__device__ inline float my_exp<float>(float x) {
    return expf(x);
}

template <>
__device__ inline double my_exp<double>(double x) {
    return exp(x);
}

// Optimized CUDA kernel with loop unrolling to reduce loop overhead.
template <typename scalar_t>
__global__ void selu_kernel_unroll(const scalar_t* __restrict__ input,
                                     scalar_t* __restrict__ output,
                                     size_t numel) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Each thread processes four elements per loop iteration using unrolling
    for (size_t i = idx; i < numel; i += stride * 4) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            size_t index = i + j * stride;
            if (index < numel) {
                scalar_t x = input[index];
                scalar_t result = (x > static_cast<scalar_t>(0))
                                  ? x
                                  : static_cast<scalar_t>(1.67326324235437728481) * (my_exp(x) - static_cast<scalar_t>(1));
                output[index] = static_cast<scalar_t>(1.05070098735548049342) * result;
            }
        }
    }
}

// Host function that launches the optimized CUDA SELU kernel
// The grid configuration accounts for the unrolling factor of 4 per thread.
torch::Tensor selu_forward(torch::Tensor input) {
    TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");

    auto output = torch::empty_like(input);
    size_t numel = input.numel();
    const int threads = 1024;
    // Adjust block count to account for 4 elements processed per thread per iteration
    const int blocks = (numel + threads * 4 - 1) / (threads * 4);

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "selu_forward_cuda_unroll", ([&] {
        const scalar_t *input_ptr = input.data_ptr<scalar_t>();
        scalar_t *output_ptr = output.data_ptr<scalar_t>();
        selu_kernel_unroll<scalar_t><<<blocks, threads>>>(input_ptr, output_ptr, numel);
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &selu_forward, "Optimized SELU Activation Forward (CUDA) with Loop Unrolling");
}
