#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32

// This kernel uses warp-level primitives to reduce branch divergence and shared memory interactions for small reductions.
__global__ void triangular_mm_kernel(const float* __restrict__ A,
                                      const float* __restrict__ B,
                                      float* __restrict__ C,
                                      int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= N || col >= N) return;

    float sum = 0.f;

    int warpRow = row & ~(WARP_SIZE - 1);
    int warpCol = col & ~(WARP_SIZE - 1);

    if (warpRow < warpCol) {
        C[row * N + col] = 0.f;
        return;
    }

    // Perform the multiplication only for the lower triangular
    if (row >= col) {
        #pragma unroll 4
        for (int k = col; k <= row; ++k) {
            sum += __ldg(&A[row * N + k]) * __ldg(&B[k * N + col]);
        }
        
        // Use warp shuffle to accumulate within the warp
        for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }

        // Only the first thread of the warp writes the result
        if (threadIdx.x % WARP_SIZE == 0) {
            C[row * N + col] = sum;
        }
    } else {
        C[row * N + col] = 0.f;
    }
}

at::Tensor forward(at::Tensor A, at::Tensor B) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2, "A must be a 2D tensor");
    TORCH_CHECK(B.dim() == 2, "B must be a 2D tensor");
    TORCH_CHECK(A.size(0) == A.size(1), "A must be square");
    TORCH_CHECK(B.size(0) == B.size(1), "B must be square");
    TORCH_CHECK(A.size(0) == B.size(0), "A and B must be the same size");

    int N = A.size(0);
    auto C = torch::empty_like(A);

    dim3 threadsPerBlock(WARP_SIZE, WARP_SIZE);
    dim3 numBlocks((N + WARP_SIZE - 1) / WARP_SIZE, (N + WARP_SIZE - 1) / WARP_SIZE);

    triangular_mm_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        N
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Triangular Matrix Multiplication using Warp Primitives (CUDA)");
}